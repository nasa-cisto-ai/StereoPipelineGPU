#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file BundleAdjustUtils.cc
///

#include <vw/Core/Log.h>
#include <vw/Camera/CameraModel.h>
#include <vw/BundleAdjustment/ControlNetwork.h>
#include <vw/Stereo/StereoModel.h>
#include <vw/Cartography/GeoReference.h>
#include <vw/FileIO/DiskImageView.h>
#include <vw/Cartography/CameraBBox.h>
#include <vw/BundleAdjustment/CameraRelation.h>
#include <asp/Core/BundleAdjustUtils.h>

#include <string>

using namespace vw;
using namespace vw::camera;
using namespace vw::stereo;
using namespace vw::ba;

namespace fs = boost::filesystem;

std::string g_piecewise_adj_str = "PIECEWISE_ADJUSTMENTS";
std::string g_session_str = "SESSION";

void asp::read_adjustments(std::string const& filename,
                           bool & piecewise_adjustments,
                           vw::Vector2 & adjustment_bounds,
                           std::vector<vw::Vector3> & position_correction,
                           std::vector<Quat> & pose_correction,
			   Vector2 & pixel_offset,
			   double & scale,
                           std::string & session) {

  // Initialize the outputs
  piecewise_adjustments = false;
  adjustment_bounds = Vector2();
  position_correction.clear();
  pose_correction.clear();
  pixel_offset = Vector2();
  scale = 1.0;
  session = "dg"; // default session, for historical reasons
  
  Vector3 pos;
  Vector4 q_buf;
  std::ifstream istr(filename.c_str());

  // Peek to see if the file contains piecewise adjustments
  std::string line;
  if (!std::getline(istr, line))
    vw_throw( ArgumentErr() << "Could not read adjustment file: " << filename << "\n" );
  if (line == g_piecewise_adj_str) {
    piecewise_adjustments = true;

    // Read the session
    std::string a, b;
    if (istr >> a >> b) {
      if (a == g_session_str) {
        session = b;
      }
    }
    
    if (! (istr >> adjustment_bounds[0] >> adjustment_bounds[1]))
      vw_throw( ArgumentErr() << "Could not read adjustment bounds from: " << filename << "\n");
  }else{
    // No piecewise adjustments. Rewind to beginning.
    piecewise_adjustments = false;
    istr.clear();
    istr.seekg(0, std::ios::beg);
  }

  // Read the actual adjustments
  while (1){
    if (! (istr >> pos[0] >> pos[1] >> pos[2]) ) break;
    if (! (istr >> q_buf[0] >> q_buf[1] >> q_buf[2] >> q_buf[3]) ) break;
    
    // The adjustments that are not piecewise may have an offset and a scale
    if (!piecewise_adjustments) {
      double a, b, c;
      if (istr >> a >> b >> c){
	pixel_offset = Vector2(a, b);
	scale = c;
      }
    }
    
    position_correction.push_back(pos);
    pose_correction.push_back(Quat(q_buf));
  }
}

// Write piecewise adjustments
void asp::write_adjustments(std::string const& filename,
                            vw::Vector2 const& adjustment_bounds,
                            std::vector<vw::Vector3> const& position_correction,
                            std::vector<vw::Quat> const& pose_correction,
                            std::string const& session) {

  std::ofstream ostr(filename.c_str());
  ostr.precision(18);

  ostr << g_piecewise_adj_str << std::endl;
  ostr << g_session_str << " " << boost::to_lower_copy(session) << std::endl;
  ostr << adjustment_bounds[0] << ' ' << adjustment_bounds[1] << std::endl;

  for (size_t adj = 0; adj < position_correction.size(); adj++) {
    ostr << position_correction[adj][0] << " "
         << position_correction[adj][1] << " "
         << position_correction[adj][2] << "\n";
    ostr << pose_correction[adj].w() << " "
         << pose_correction[adj].x() << " "
         << pose_correction[adj].y() << " "
         << pose_correction[adj].z() << " " << "\n";
  }
  ostr.close();
}

void asp::write_adjustments(std::string const& filename,
                       Vector3 const& position_correction,
                       Quat const& pose_correction) {
  std::ofstream ostr(filename.c_str());
  ostr.precision(18);
  ostr << position_correction[0] << " " << position_correction[1] << " "
       << position_correction[2] << "\n";
  ostr << pose_correction.w() << " " << pose_correction.x() << " "
       << pose_correction.y() << " " << pose_correction.z() << " " << "\n";
  ostr.close();
}

void asp::compute_stereo_residuals(std::vector<boost::shared_ptr<CameraModel>> const& camera_models,
                                   ControlNetwork const& cnet) {

  // Compute pre-adjustment residuals and convert to bundles
  int n = 0;
  double error_sum = 0;
  double min_error = ScalarTypeLimits<double>::highest();
  double max_error = ScalarTypeLimits<double>::lowest();
  for (size_t i = 0; i < cnet.size(); ++i) {
    for (size_t j = 0; j+1 < cnet[i].size(); ++j) {
      ++n;
      size_t cam1 = cnet[i][j].image_id();
      size_t cam2 = cnet[i][j+1].image_id();
      Vector2 pix1 = cnet[i][j].position();
      Vector2 pix2 = cnet[i][j+1].position();

      StereoModel sm(camera_models[cam1].get(), camera_models[cam2].get());
      double error;
      sm(pix1,pix2,error);
      error_sum += error;
      min_error = std::min(min_error, error);
      max_error = std::max(max_error, error);
    }
  }
  vw_out() << "\nStereo intersection residuals -- min: " << min_error
           << "  max: " << max_error << "  average: " << (error_sum/n) << "\n";
}

// See the .h file for documentation
vw::BBox2 asp::camera_bbox_with_cache(std::string const& dem_file,
                                      std::string const& image_file,
                                      boost::shared_ptr<vw::camera::CameraModel> const&
                                      camera_model,
                                      std::string const& out_prefix) {
  
  namespace fs = boost::filesystem;

  vw_out() << "Computing ground footprint bounding box of: " + image_file << std::endl;

  vw::BBox2 box;
  
  std::string box_path = out_prefix + '-' + fs::path(image_file).stem().string() + "-bbox.txt";
  if (fs::exists(box_path)) {
    double min_x, min_y, max_x, max_y;
    std::ifstream ifs(box_path);
    if (ifs >> min_x >> min_y >> max_x >> max_y) {
      box.min() = vw::Vector2(min_x, min_y);
      box.max() = vw::Vector2(max_x, max_y);
      vw_out() << "Read cached ground footprint bbox from: " << box_path << ":\n" << box << "\n";
      return box;
    }
  }

  // Read the DEM and supporting data
  vw::cartography::GeoReference dem_georef;
  DiskImageView<float> dem_disk_image(dem_file);
  ImageViewRef<PixelMask<float>> dem;
  boost::shared_ptr<DiskImageResource> dem_rsrc(DiskImageResourcePtr(dem_file));
  if (dem_rsrc->has_nodata_read())
    dem = create_mask(dem_disk_image, dem_rsrc->nodata_read());
  else
    dem = pixel_cast<PixelMask<float>>(dem_disk_image); // all pixels are valid
  
  bool has_georef = vw::cartography::read_georeference(dem_georef, dem_file);
  if (!has_georef)
    vw_throw( ArgumentErr() << "There is no georeference information in: "
              << dem_file << ".\n" );

  try {
    DiskImageView<float> img(image_file);
    float auto_res = -1.0;  // Will be updated
    bool quick = false;     // Do a thorough job
    box = vw::cartography::camera_bbox(dem, dem_georef, dem_georef,
                                       camera_model, img.cols(), img.rows(),
                                       auto_res, quick);
  } catch (std::exception const& e) {
    vw_throw( ArgumentErr() << e.what() << "\n"
              << "Failed to compute the footprint of camera image: " << image_file
              << " onto DEM: " << dem_file << ".\n");
  }

  vw_out() << "Writing: " << box_path << "\n";
  std::ofstream ofs(box_path.c_str());
  ofs.precision(17);
  ofs << box.min().x() << " " <<  box.min().y() << " "
      << box.max().x() << " " <<  box.max().y() << "\n";
  ofs.close();
  
  return box;
}

// See the .h file for the documentation.
void asp::build_overlap_list_based_on_dem
/*        */ (std::string const& out_prefix, std::string const& dem_file, double pct_for_overlap,
              std::vector<std::string> const& image_files,
              std::vector<boost::shared_ptr<vw::camera::CameraModel>> const& camera_models,
              std::set<std::pair<std::string, std::string>> & overlap_list) {

  // Wipe the output
  overlap_list.clear();
  
  // Sanity check
  if (image_files.size() != camera_models.size())
    vw_throw( ArgumentErr() << "Expecting as many images as cameras.\n");
  
  int num_images = image_files.size();
  std::vector<vw::BBox2> boxes(num_images);
  for (int it = 0; it < num_images; it++) {
    // By this stage the camera bboxes are already computed and cached,
    // they just need to be loaded.
    boxes[it] = asp::camera_bbox_with_cache(dem_file, image_files[it], camera_models[it],  
                                            out_prefix);

    // Expand the box by the given factor
    double factor = pct_for_overlap / 100.0;

    // The expansion factor can be negative, but not if it results in an empty box
    if (factor <= -1.0) 
      vw_throw(ArgumentErr() << "Invalid percentage when computing the footprint of camera image: "
               << pct_for_overlap  << ".\n");
      
    double half_extra_x = 0.5 * boxes[it].width()  * factor;
    double half_extra_y = 0.5 * boxes[it].height() * factor;
    boxes[it].min() -= Vector2(half_extra_x, half_extra_y);
    boxes[it].max() += Vector2(half_extra_x, half_extra_y);
  }

  // See which boxes overlap. While this is an O(N^2) computation,
  // likely N is at most a thousand or two, which should be
  // manageable. A 2D tree of box corners could be used, and two boxes
  // would then overlap if corners from one box are contained in a
  // second box. That would be a O(N * log(N)) lookup.
  // TODO(oalexan1): Use a tree.
  for (int it1 = 0; it1 < num_images; it1++) {
    for (int it2 = it1 + 1; it2 < num_images; it2++) {
      BBox2 box = boxes[it1]; // deep copy
      box.crop(boxes[it2]);
      if (!box.empty())
        overlap_list.insert(std::make_pair(image_files[it1], image_files[it2]));
    }
  }

  return;
}

// Convert dir1/image1.cub or dir1/image1.xml to out-prefix-image1.adjust
std::string asp::bundle_adjust_file_name(std::string const& prefix,
                                         std::string const& input_img,
                                         std::string const& input_cam){

  // Create the adjusted camera file name from the original camera filename,
  // unless it is empty, and then use the image file name.
  std::string file = input_cam;
  if (file == "")
    file = input_img;

  return prefix + "-" + fs::path(file).stem().string() + ".adjust";
}

/// Ensure that no images, camera files, or adjustment names are duplicate.
/// That will cause the output files to overwrite each other!
void asp::check_for_duplicates(std::vector<std::string> const& image_files,
                               std::vector<std::string> const& camera_files,
                               std::string const& out_prefix) {

  if (image_files.size() != camera_files.size())
    vw_throw(vw::ArgumentErr() << "Expecting as many images as cameras.\n");
  
  std::set<std::string> img_set, cam_set, adj_set;
  for (size_t i = 0; i < camera_files.size(); i++) {

    std::string const & img = image_files[i];  // alias
    std::string const & cam = camera_files[i]; // alias
    std::string         adj = asp::bundle_adjust_file_name(out_prefix, img, cam);

    if (img_set.find(img) != img_set.end()) 
      vw_throw(vw::ArgumentErr() << "Found duplicate image: " << img << "\n");
    
    if (cam != "" && cam_set.find(cam) != cam_set.end()) 
      vw_throw(vw::ArgumentErr() << "Found duplicate camera: " << cam << "\n");

    if (adj_set.find(adj) != adj_set.end()) 
      vw_throw(vw::ArgumentErr() << "Found duplicate adjustment name: " << adj << "\n");

    img_set.insert(img);
    if (cam != "") cam_set.insert(cam);
    adj_set.insert(adj);
    
  }
}

// Make a list of all of the image pairs to find matches for
void asp::determine_image_pairs(// Inputs
                                int overlap_limit,
                                bool match_first_to_last,
                                std::vector<std::string> const& image_files,
                                // if having optional preexisting camera positions
                                bool got_est_cam_positions,
                                // Optional filter distance, set to -1 if not used
                                double position_filter_dist,
                                // Estimated camera positions, set to empty if missing
                                std::vector<vw::Vector3> const& estimated_camera_gcc,
                                // Optional preexisting list
                                bool have_overlap_list,
                                std::set<std::pair<std::string, std::string>> const&
                                overlap_list,
                                // Output
                                std::vector<std::pair<int,int>> & all_pairs) {

  // Wipe the output
  all_pairs.clear();

  // Need this to avoid repetitions
  std::set<std::pair<int, int>> local_set;
  
  int num_images = image_files.size();
  for (int i0 = 0; i0 < num_images; i0++){

    for (int j0 = i0 + 1; j0 <= i0 + overlap_limit; j0++){

      // Make copies of i and j which we can modify
      int i = i0, j = j0;

      if (j >= num_images) {
        
        if (!match_first_to_last)
          continue; // out of bounds

        j = j % num_images; // wrap around

        if (i == j) 
          continue; // can't have matches to itself

        if (i > j) 
          std::swap(i, j);
      }
      
      // Apply the overlap list if manually specified. Otherwise every
      // image pair i, j as above will be matched.
      if (have_overlap_list) {
        auto pair1 = std::make_pair(image_files[i], image_files[j]);
        auto pair2 = std::make_pair(image_files[j], image_files[i]);
        if (overlap_list.find(pair1) == overlap_list.end() &&
            overlap_list.find(pair2) == overlap_list.end())
          continue;
      }

      // If this option is set, don't try to match cameras that are too far apart.
      if (got_est_cam_positions && (position_filter_dist > 0)) {
        Vector3 this_pos  = estimated_camera_gcc[i];
        Vector3 other_pos = estimated_camera_gcc[j];
        if ((this_pos  != Vector3(0,0,0)) && // If both positions are known
            (other_pos != Vector3(0,0,0)) && // and they are too far apart
            (norm_2(this_pos - other_pos) > position_filter_dist)) {
          vw_out() << "Skipping position: " << this_pos << " and "
                   << other_pos << " with distance " << norm_2(this_pos - other_pos)
                   << std::endl;
          continue; // Skip this image pair
        }
      }

      local_set.insert(std::make_pair(i,j));
    }
  }

  // The pairs without repetition
  for (auto it = local_set.begin(); it != local_set.end(); it++)
    all_pairs.push_back(*it);
}

/// Load a DEM from disk to use for interpolation.
void asp::create_interp_dem(std::string const& dem_file,
                       vw::cartography::GeoReference & dem_georef,
                       ImageViewRef<PixelMask<double>> & interp_dem){
  
  vw_out() << "Loading DEM: " << dem_file << std::endl;

  // Read the no-data
  double nodata_val = -std::numeric_limits<float>::max(); // note we use a float nodata
  if (vw::read_nodata_val(dem_file, nodata_val))
    vw_out() << "Found DEM nodata value: " << nodata_val << std::endl;

  // Create the interpolated DEM
  ImageViewRef<PixelMask<double>> dem
    = create_mask(DiskImageView<double>(dem_file), nodata_val);
  interp_dem = interpolate(dem, BilinearInterpolation(), ConstantEdgeExtension());

  // Read the georef
  bool is_good = vw::cartography::read_georeference(dem_georef, dem_file);
  if (!is_good) {
    vw_throw(ArgumentErr() << "Error: Cannot read a georeference from DEM: "
             << dem_file << ".\n");
  }
}

// Given an xyz point in ECEF coordinates, update its height above datum
// by interpolating into a DEM. The user must check the return status.
bool asp::update_point_height_from_dem(vw::cartography::GeoReference const& dem_georef,
                                       vw::ImageViewRef<PixelMask<double>> const& interp_dem,
                                       // Output
                                       vw::Vector3 & xyz) {

  // Points at planet center are outliers
  if (xyz == Vector3(0, 0, 0))
    return false;
  
  Vector3 llh = dem_georef.datum().cartesian_to_geodetic(xyz);
  Vector2 ll  = subvector(llh, 0, 2);
  Vector2 pix = dem_georef.lonlat_to_pixel(ll);
  if (!interp_dem.pixel_in_bounds(pix))
    return false;

  PixelMask<double> height = interp_dem(pix[0], pix[1]);
  if (!is_valid(height))
    return false;
  
  llh[2] = height.child();

  // NaN check
  if (llh[2] != llh[2]) 
    return false;

  // Overwrite the input
  xyz = dem_georef.datum().geodetic_to_cartesian(llh);

  return true;
}

/// Try to update the elevation of a GCC coordinate from a DEM, overwriting
// the array 'point'.
/// - Returns false if the point falls outside the DEM or in a hole.
bool asp::update_point_height_from_dem(double* point,
                                       vw::cartography::GeoReference const& dem_georef,
                                       vw::ImageViewRef<vw::PixelMask<double>> const& interp_dem) {

  Vector3 xyz(point[0], point[1], point[2]);

  if (!update_point_height_from_dem(dem_georef, interp_dem,
                                    // Output
                                    xyz))
    return false;

  // Overwrite the input
  for (size_t it = 0; it < xyz.size(); it++) 
    point[it] = xyz[it];
  
  return true;
}

// Given a set of xyz points and a DEM to interpolate into,
// create a vector of xyz points which are updated to be at the height given
// by the DEM. This assumes that interp_dem is created outside of this
// function with bilinear interpolation, via asp::create_interp_dem().
// Invalid or uncomputable xyz are set to the zero vector.
void asp::update_point_height_from_dem(vw::ba::ControlNetwork const& cnet,
                                       std::set<int> const& outliers,
                                       vw::cartography::GeoReference const& dem_georef,
                                       vw::ImageViewRef<vw::PixelMask<double>> const& interp_dem,
                                       // Output
                                       std::vector<vw::Vector3> & dem_xyz_vec) {

  // Initialize the output
  int num_tri_points = cnet.size();
  dem_xyz_vec = std::vector<vw::Vector3>(num_tri_points, vw::Vector3(0, 0, 0));
  
  for (int ipt = 0; ipt < num_tri_points; ipt++) {
    
    if (cnet[ipt].type() == vw::ba::ControlPoint::GroundControlPoint)
      continue; // GCP keep their own thing
    
    if (outliers.find(ipt) != outliers.end())
      continue; // Skip outliers
    
    Vector3 observation = cnet[ipt].position(); // will get overwritten
    if (asp::update_point_height_from_dem(dem_georef, interp_dem,  
                                          // Output
                                          observation))
      dem_xyz_vec[ipt] = observation;
  }

  return;
}

// Shoot rays from all matching interest point. Intersect those with a
// DEM. Find their average.  Invalid or uncomputable xyz are set to
// the zero vector.
void asp::calc_avg_intersection_with_dem(vw::ba::ControlNetwork const& cnet,
                                         vw::ba::CameraRelationNetwork<vw::ba::JFeature> const& crn,
                                         std::set<int> const& outliers,
                                         std::vector<boost::shared_ptr<vw::camera::CameraModel>>
                                         const& camera_models,
                                         vw::cartography::GeoReference const& dem_georef,
                                         vw::ImageViewRef<vw::PixelMask<double>> const& interp_dem,
                                         // Output
                                         std::vector<vw::Vector3> & dem_xyz_vec) {
  
  int num_tri_points = cnet.size();

  dem_xyz_vec = std::vector<vw::Vector3>(num_tri_points, vw::Vector3(0, 0, 0));
  std::vector<int> dem_xyz_count(num_tri_points, 0);
  
  for (int icam = 0; icam < (int)crn.size(); icam++) {
    
    for (auto fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++) {
        
      // The index of the 3D point
      int ipt = (**fiter).m_point_id;

      if (cnet[ipt].type() == vw::ba::ControlPoint::GroundControlPoint)
        continue; // GCP do not get modified
      
      if (outliers.find(ipt) != outliers.end())
        continue; // Skip outliers
        
      // The observed value for the projection of point with index ipt into
      // the camera with index icam.
      Vector2 observation = (**fiter).m_location;
        
      // Ideally this point projects back to the pixel observation, so use the
      // triangulated position as initial guess.
      Vector3 xyz_guess = cnet[ipt].position();

      // Points at planet center are outliers. This check is likely redundant,
      // but good to have.
      if (xyz_guess == Vector3(0, 0, 0))
        continue;

      bool treat_nodata_as_zero = false;
      bool has_intersection = false;
      double height_error_tol = 0.001; // 1 mm should be enough
      double max_abs_tol      = 1e-14; // abs cost fun change b/w iterations
      double max_rel_tol      = 1e-14;
      int num_max_iter        = 25;   // Using many iterations can be very slow

      Vector3 dem_xyz = vw::cartography::camera_pixel_to_dem_xyz
        (camera_models[icam]->camera_center(observation),
         camera_models[icam]->pixel_to_vector(observation),
         interp_dem, dem_georef, treat_nodata_as_zero, has_intersection,
         height_error_tol, max_abs_tol, max_rel_tol, num_max_iter, xyz_guess);

      if (!has_intersection) 
        continue;

      dem_xyz_vec[ipt] += dem_xyz;
      dem_xyz_count[ipt]++;
    }
  }

  // Average the successful intersections
  for (size_t xyz_it = 0; xyz_it < dem_xyz_vec.size(); xyz_it++) {
    if (dem_xyz_count[xyz_it] > 0) 
      dem_xyz_vec[xyz_it] = dem_xyz_vec[xyz_it] / double(dem_xyz_count[xyz_it]);
    else
      dem_xyz_vec[xyz_it] = Vector3();
  }

  return;
}

// Flag outliers by reprojection error with input cameras. This assumes that
// the input cameras are pretty accurate.
void asp::flag_initial_outliers(vw::ba::ControlNetwork const& cnet,
                                vw::ba::CameraRelationNetwork<vw::ba::JFeature> const& crn,
                                std::vector<boost::shared_ptr<vw::camera::CameraModel>>
                                const& camera_models,
                                double max_init_reproj_error,
                                // Output
                                std::set<int> & outliers) {
  // Wipe the output
  outliers.clear();

  int num_cameras = camera_models.size();
  int num_tri_points = cnet.size();

  for (int icam = 0; icam < (int)crn.size(); icam++) {

    for (auto fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++) {
      
      // The index of the triangulated point
      int ipt = (**fiter).m_point_id;
      
      VW_ASSERT(icam < num_cameras, ArgumentErr() << "Out of bounds in the number of cameras.");
      VW_ASSERT(ipt < num_tri_points, ArgumentErr() << "Out of bounds in the number of points.");

      if (outliers.find(ipt) != outliers.end()) {
        // Is an outlier
        continue;
      }
      
      // The observed value for the projection of point with index ipt into
      // the camera with index icam.
      Vector2 observation = (**fiter).m_location;

      Vector3 const& tri_point = cnet[ipt].position(); // alias

      if (tri_point == Vector3(0, 0, 0)) {
        // Points at planet center are outliers
        outliers.insert(ipt);
        continue;
      }
      
      vw::Vector2 pix;
      try {
        pix = camera_models[icam]->point_to_pixel(tri_point);
        bool is_good = (norm_2(pix - observation) <= max_init_reproj_error);
        if (!is_good) { // this checks for NaN too
          outliers.insert(ipt);
          continue;
        }
      } catch(...) {
        outliers.insert(ipt);
        continue;
      }
    }
    
  } // end iterating over cameras

  return;
}

// Manufacture a CSM state file from an adjust file
std::string asp::csmStateFile(std::string const& adjustFile) {

  std::string csmFile = adjustFile;
  
  // If the suffix we want to add is already present, remove it first
  std::string suff = ".adjusted_state";
  auto it = csmFile.find(suff);
  if (it != std::string::npos)
    csmFile.replace(it, suff.size(), "");
  
  csmFile = boost::filesystem::path(csmFile).replace_extension(suff + ".json").string();

  return csmFile;
}

  
