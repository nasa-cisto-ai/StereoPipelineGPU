#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Core/OutlierProcessing.h>

#include <vw/Image/Statistics.h>
#include <vw/Math/Statistics.h>
#include <vw/Core/Stopwatch.h>

using namespace vw;

namespace asp {

// Estimate a bounding box without outliers. Note that individual percentage factors
// are used in x, y, and z. These are supposed to around 0.75 or so. The outlier factor is 3.0
// or so.
void estimate_inliers_bbox(double pct_factor_x, double pct_factor_y, double pct_factor_z,
                           double outlier_factor,
                           std::vector<double> const& x_vals,
                           std::vector<double> const& y_vals,
                           std::vector<double> const& z_vals,
                           vw::BBox3 & inliers_bbox) {
  
  // Initialize the output
  inliers_bbox = BBox3();

  double bx, ex, by, ey, bz, ez;
  if (!vw::math::find_outlier_brackets(x_vals, pct_factor_x, outlier_factor, bx, ex))
    return;
  if (!vw::math::find_outlier_brackets(y_vals, pct_factor_y, outlier_factor, by, ey))
    return;
  if (!vw::math::find_outlier_brackets(z_vals, pct_factor_z, outlier_factor, bz, ez))
    return;

  // NaN values will result in an error further down
  if (ex != ex || ey != ey || ez != ez)
    return;  

  // Need to compute the next double because the VW bounding box is
  // exclusive at the top.
  ex = boost::math::nextafter(ex, std::numeric_limits<double>::max());
  ey = boost::math::nextafter(ey, std::numeric_limits<double>::max());
  ez = boost::math::nextafter(ez, std::numeric_limits<double>::max());

  inliers_bbox.grow(Vector3(bx, by, bz));
  inliers_bbox.grow(Vector3(ex, ey, ez));
}

// Get a generous estimate of the bounding box of the current set
// while excluding outliers
void estimate_points_bdbox(vw::ImageViewRef<vw::Vector3> const& proj_points,
                           vw::ImageViewRef<double> const& error_image,
                           vw::Vector2 const& remove_outliers_params,
                           double estim_max_error,
                           vw::BBox3 & inliers_bbox) {

  // TODO(oalexan1): Here it may help to do several passes. First throw out the worst
  // outliers, then estimate the box from the remaining points, etc.
  
  std::vector<double> x_vals, y_vals, z_vals;
  for (int col = 0; col < proj_points.cols(); col++){
    for (int row = 0; row < proj_points.rows(); row++){

      // Avoid points marked as not valid
      Vector3 P = proj_points(col, row);
      if (P != P)
        continue;

      // Make use of the estimated error, if available
      if (estim_max_error > 0 && error_image(col, row) > estim_max_error) 
        continue;

      x_vals.push_back(P.x());
      y_vals.push_back(P.y());
      z_vals.push_back(P.z());
    }
  }

  double pct_factor     = remove_outliers_params[0]/100.0; // e.g., 0.75
  double outlier_factor = remove_outliers_params[1];       // e.g., 3.0.

  // Make these more generous, as we want to throw out only the worst
  // outliers. Note that we are even more generous in z, to avoid cutting
  // of isolated mountain peaks. This is a bugfix.
  double pct_factor_x = (1.0 + pct_factor)/2.0; // e.g., 0.875 
  double pct_factor_y = (1.0 + pct_factor)/2.0; // e.g., 0.875 
  double pct_factor_z = (3.0 + pct_factor)/4.0; // e.g., 0.9375

  // Double this factor, now it will equal 6.  With a small factor, if
  // the domain of the DEM is a rectangle rotated by 45 degrees, it
  // may cut off corners.
  outlier_factor *= 2.0;

  // Call auxiliary function to do the estimation
  estimate_inliers_bbox(pct_factor_x, pct_factor_y, pct_factor_z, outlier_factor, 
                       x_vals, y_vals, z_vals, inliers_bbox);
  
  return;
}

// A class to pick some samples to estimate the range of values
// of a given dataset
class ErrorRangeEstimAccum: public ReturnFixedType<void> {
  typedef double accum_type;
  std::vector<accum_type> m_vals;
public:
  typedef accum_type value_type;
  
  ErrorRangeEstimAccum() { m_vals.clear(); }
  
  void operator()( accum_type const& value ) {
    // Don't add zero errors, those most likely came from invalid points
    if (value > 0)
      m_vals.push_back(value);
  }
  
  int size(){
    return m_vals.size();
  }
  
  value_type value(Vector2 const& remove_outliers_params){
    VW_ASSERT(!m_vals.empty(), ArgumentErr() << "ErrorRangeEstimAccum: no valid samples");
    
    // How to pick a representative value for maximum error?  The
    // maximum error itself may be no good, as it could be very
    // huge, and then sampling the range of errors will be distorted
    // by that.  The solution adopted here: Find a percentile of the
    // range of errors, mulitply it by the outlier factor, and
    // multiply by another factor to ensure we don't underestimate
    // the maximum. This value may end up being larger than the
    // largest error, but at least it is is not grossly huge
    // if just a few of the errors are very large.
    std::sort(m_vals.begin(), m_vals.end());
    int    len    = m_vals.size();
    double pct    = remove_outliers_params[0]/100.0; // e.g., 0.75
    double factor = remove_outliers_params[1];
    int    k      = std::min(len - 1, (int)(pct*len));
    
    // Care here with empty sets
      if (k >= 0) 
        return m_vals[k]*factor*4.0;
      
      return 0;
  }
  
};
  
// Sample the image and get generous estimates (but without outliers)
// of the maximum triangulation error and of the 3D box containing the
// projected points. These will be tightened later.
double estim_max_tri_error_and_proj_box(vw::ImageViewRef<vw::Vector3> const& proj_points,
                                        vw::ImageViewRef<double> const& error_image,
                                        vw::Vector2 const& remove_outliers_params,
                                        vw::BBox3 & estim_proj_box) {

  // Initialize the outputs
  double estim_max_error = 0.0;
  estim_proj_box = BBox3();

  // Start with a 256 (2^8) by 256 sampling of the cloud
  bool success = false;
  for (int attempt = 8; attempt <= 18; attempt++){
    
    double sample = (1 << attempt);
    int32 subsample_amt = int32(norm_2(Vector2(error_image.cols(), error_image.rows()))/sample);
    if (subsample_amt < 1 )
      subsample_amt = 1;
    
    Stopwatch sw2;
    sw2.start();
    PixelAccumulator<asp::ErrorRangeEstimAccum> error_accum;
    for_each_pixel(subsample(error_image, subsample_amt),
                   error_accum,
                   TerminalProgressCallback
                   ("asp","Bounding box and triangulation error range estimation: ") );
    if (error_accum.size() > 0){
      success = true;
      estim_max_error = error_accum.value(remove_outliers_params);
    }
    sw2.stop();

    asp::estimate_points_bdbox(subsample(proj_points, subsample_amt),
                               subsample(error_image, subsample_amt),
                               remove_outliers_params,  estim_max_error,
                               estim_proj_box);
    
    if (estim_proj_box.empty()) 
      success = false;
    
    vw_out(DebugMessage,"asp") << "Elapsed time: " << sw2.elapsed_seconds() << std::endl;
    if (success || subsample_amt == 1) break;
    vw_out() << "Estimation failed. Check if your cloud is valid. "
             << "Trying again with finer sampling.\n";
  }
  return estim_max_error;
}
  
} // end namespace asp
