#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <vw/Core/Exception.h>
#include <asp/Core/OpenCVUtils.h>

#include <Eigen/Core>
#include <opencv2/core/eigen.hpp>

#include <iostream>

namespace asp {

  // Return the OpenCV matrix type as a string
  std::string cvMatType(cv::Mat const& M) {

    int inttype = M.type();

    std::string r, a;
    
    uchar depth = inttype & CV_MAT_DEPTH_MASK;
    uchar chans = 1 + (inttype >> CV_CN_SHIFT);

    switch (depth) {
    case CV_8U:  r = "8U"; break;
    case CV_8S:  r = "8S"; break;
    case CV_16U: r = "16U"; break;
    case CV_16S: r = "16S"; break;
    case CV_32S: r = "32S"; break;
    case CV_32F: r = "32F"; break;
    case CV_64F: r = "64F"; break;
    default:     r = "User"; break;
    }   
    r += "C";
    r += (chans+'0');

    return r;
  }

  // Convert a matrix from OpenCV to the VisionWorkbench matrix type.
  // Implemented only for double precision matrix (should be easy to
  // extend also for single precision and various integer types).
  vw::Matrix<double> cvMatToVwMat(cv::Mat const& M) {

    std::string matType = asp::cvMatType(M);
    if (matType != "64FC1") {
      vw::vw_throw(vw::NoImplErr() << "cvMatToVwMat() implemented only for "
                   << "double-precision OpenCV matrices (64FC1). Got: " << matType);
      
    }
    
    Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic> EM;
    cv::cv2eigen(M, EM);

    vw::Matrix<double> VM(EM.rows(), EM.cols());

    for (int row = 0; row < EM.rows(); row++) {
      for (int col = 0; col < EM.cols(); col++) {
        VM(row, col) = EM(row, col);
      }
    }

    return VM;
  }
  

  // Scale an image from [0, 1] to [0, 255], round, and clamp.
  // NaN values are not affected.
  void formScaledByteCVImage(vw::ImageViewRef<float> in, cv::Mat & out) {
    
    out = cv::Mat(in.rows(), in.cols(), CV_8UC1, cv::Scalar(0));

    // Note how we read from in(col, row) but write to out(row, col)
    
    for (int row = 0; row < in.rows(); row++) {
      for (int col = 0; col < in.cols(); col++) {
        int val = round(255.0 * in(col, row));
        if (val < 0) 
          val = 0;
        if (val > 255)
          val = 255;
      
        out.at<uint8_t>(row, col) = val;
      }
    }

    return;
  }

  // Insert an image as a block at a desired location in a bigger image
  void cvInsertBlock(cv::Mat const& input_image, int extra_x,
                     int extra_y, cv::Mat& output_image) {
    input_image.copyTo(output_image(cv::Range(extra_y, extra_y + input_image.rows),
                                    cv::Range(extra_x, extra_x + input_image.cols)));
  }
  
} // end namespace asp
