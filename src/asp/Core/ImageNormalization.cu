#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/FileIO/DiskImageResource.h>
#include <vw/Core/Log.h>

#include <asp/Core/ImageNormalization.h>
#include <asp/Core/StereoSettings.h>

#include <limits>

using namespace vw;

namespace asp {
  
  /// Returns the correct nodata value from the input images or the input options
  void get_nodata_values(boost::shared_ptr<vw::DiskImageResource> left_rsrc,
                         boost::shared_ptr<vw::DiskImageResource> right_rsrc,
                         float & left_nodata_value,
                         float & right_nodata_value){
    
    // The no-data value read from options overrides the value present in the image files.
    left_nodata_value  = std::numeric_limits<float>::quiet_NaN();
    right_nodata_value = std::numeric_limits<float>::quiet_NaN();
    if (left_rsrc->has_nodata_read ()) left_nodata_value  = left_rsrc->nodata_read();
    if (right_rsrc->has_nodata_read()) right_nodata_value = right_rsrc->nodata_read();
    
    float user_nodata = stereo_settings().nodata_value;
    if (!std::isnan(user_nodata)){
      
      if (user_nodata < left_nodata_value)
        vw_out(WarningMessage) << "It appears that the user-supplied no-data value is less than "
                               << "the no-data value of left image. This may not be what was "
                               << "intended.\n";
      if (user_nodata < right_nodata_value)
        vw_out(WarningMessage) << "It appears that the user-supplied no-data value is less than "
                               << "the no-data value of right image. This may not be what was "
                               << "intended.\n";

      left_nodata_value  = user_nodata;
      right_nodata_value = user_nodata;
    }

    return;
  }
  
}
