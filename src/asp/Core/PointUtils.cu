#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// TODO(oalexan1): Move all LAS, PCD, and CSV logic to its own
// PointReader.cc class, as this file is too big and very slow to compile.

/// \file PointUtils.cc
///

// Turn off warnings about things we can't control
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations"
#include <liblas/liblas.hpp>
#pragma GCC diagnostic pop

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
#include <asp/Core/PointUtils.h>
#include <vw/Cartography/Chipper.h>
#include <vw/Core/Stopwatch.h>
#include <boost/math/special_functions/fpclassify.hpp>
#include <boost/math/special_functions/next.hpp>

using namespace vw;
using namespace vw::cartography;
using namespace pdal::filters;


namespace asp{

  class LasReader: public BaseReader{
    liblas::Reader& m_reader;
  public:

    LasReader(liblas::Reader & reader):m_reader(reader){
      liblas::Header const& header = m_reader.GetHeader();
      m_num_points = header.GetPointRecordsCount();

      std::string wkt = header.GetSRS().GetWKT();
      m_has_georef = false;
      if (wkt != ""){
        m_has_georef = true;
        m_georef.set_wkt(wkt);
      }
    }

    virtual bool ReadNextPoint(){
      return m_reader.ReadNextPoint();
    }

    virtual Vector3 GetPoint(){
      liblas::Point const& p = m_reader.GetPoint();
      return Vector3(p.GetX(), p.GetY(), p.GetZ());
    }

  };

  class CsvReader: public BaseReader{
    std::string  m_csv_file;
    asp::CsvConv m_csv_conv;
    bool         m_is_first_line;
    bool         m_has_valid_point;
    Vector3      m_curr_point;
    std::ifstream * m_ifs;
  public:

    CsvReader(std::string const & csv_file,
              asp::CsvConv const& csv_conv,
              GeoReference const& georef)
      : m_csv_file(csv_file), m_csv_conv(csv_conv),
        m_is_first_line(true), m_has_valid_point(false) {

      // We will convert from projected space to xyz, unless points
      // are already in this format.
      m_has_georef = (m_csv_conv.format != asp::CsvConv::XYZ);

      m_georef      = georef;
      m_num_points  = asp::csv_file_size(m_csv_file);

      m_ifs = new std::ifstream ( m_csv_file.c_str() );
      if ( !*m_ifs ) {
        vw_throw( vw::IOErr() << "Unable to open file \"" << m_csv_file << "\"" );
      }

      VW_ASSERT(m_csv_conv.csv_format_str != "",
                ArgumentErr() << "CsvReader: The CSV format was not specified.\n");

    }

    virtual bool ReadNextPoint() {

      std::string line;
      asp::CsvConv::CsvRecord vals;

      // Keep on reading, until a valid point is hit or the end of the file
      // is reached.
      while (1) {
        m_has_valid_point = static_cast<bool>(getline(*m_ifs, line, '\n'));
        if (!m_has_valid_point) return m_has_valid_point; // reached end of file

        vals = m_csv_conv.parse_csv_line(m_is_first_line, m_has_valid_point, line);
        if (m_has_valid_point) break;
      }

      // Will return projected point and height or xyz. We really
      // prefer projected points, as then the chipper will have an
      // easier time grouping spatially points close together, as it
      // operates the first two coordinates.
      bool return_point_height = true;
      m_curr_point = m_csv_conv.csv_to_cartesian_or_point_height
        (vals, m_georef, return_point_height);

      return m_has_valid_point;
    }

    virtual Vector3 GetPoint(){
      return m_curr_point;
    }

    virtual ~CsvReader(){
      delete m_ifs;
      m_ifs = NULL;
    }

  }; // End class CsvReader

  void PcdReader::read_header() {
    // Open the file as text
    std::ifstream handle;
    handle.open(m_pcd_file.c_str());
    if (handle.fail()) {
      vw_throw( vw::IOErr() << "Unable to open file \"" << m_pcd_file << "\"" );
    }
    // Start checking all of the header elements
    bool valid = true;
    std::string line, dummy, value;
    std::getline(handle, line);     
    while (line[0] == '#') // Skip initial comment lines
      std::getline(handle, line);
    // Check the header version - we only support one kind for now.
    boost::to_lower(line);
    if (line.find("version 0.7") == std::string::npos) {
      vw_out() << "Error: Unsupported PCD file version: " << line << std::endl;
      valid = false;
    }
    // Verify the fields
    std::getline(handle, line);
    boost::to_lower(line);
    if (line.find("fields x y z") == std::string::npos) {
      vw_out() << "Error: Unsupported PCD fields: " << line << std::endl;
      valid = false;
    }
    // Get some other information, no checking here...
    handle >> dummy >> m_size_bytes;
    std::getline(handle, line); // Go to the next line
    if ((m_size_bytes != 4) && (m_size_bytes != 8)) {
      vw_out() << "Error: Unsupported byte size: " << m_size_bytes << std::endl;
      valid = false;
    }
    handle >> dummy >> m_type;
    std::getline(handle, line); // Go to the next line
    if (m_type == 'F')
      m_type = 'f';
    if (m_type != 'f') {
      vw_out() << "Error: Currently only Float type PCD files are supported!\n";
      valid = false;
    }
    
    // Get size info
    int width, height, count;
    handle >> dummy >> count;
    std::getline(handle, line); // Go to the next line
    if (count != 1) {
      vw_out() << "Error: Unsupported PCD count: " << count << std::endl;
      valid = false;
    }
    handle >> dummy >> width >> dummy >> height;
    std::getline(handle, line); // Skip viewpoint line
    std::getline(handle, line);
    handle >> dummy >> m_num_points;
    if (m_num_points != (static_cast<size_t>(width*height))) {
      vw_out() << "Error: PCD point count error!\n";
      valid = false;
    }
    // Get the type of file, ascii or binary
    handle >> dummy >> value;
    boost::to_lower(value);
    m_binary_format = (value != "ascii");
    
    if (handle.fail()) {
      vw_out() << "Error: PCD read error!\n";
      valid = false;
    }
    
    m_header_length_bytes = handle.tellg();
    
    // Stop reading the header file
    handle.close();
    if (!valid)
      vw_throw(ArgumentErr() << "Fatal error reading PCD file: " << m_pcd_file);
  }

  
  PcdReader::PcdReader(std::string const & pcd_file)
    : m_pcd_file(pcd_file), m_has_valid_point(false){

    // For now PCD files are required to be in XYZ GCC format.
    m_has_georef = false;

    read_header();      
    
    // Open the file for data reading in the proper format then skip past the header
    if (m_binary_format)
      m_ifs = new std::ifstream ( m_pcd_file.c_str(), std::ios_base::binary);
    else
      m_ifs = new std::ifstream ( m_pcd_file.c_str());

    m_ifs->seekg(m_header_length_bytes);
  }

  bool PcdReader::ReadNextPoint(){

    // Check if there is more data
    if (!m_ifs->good()) {
      m_has_valid_point = false;
      return false;
    }

    if (m_binary_format) {

      if (m_size_bytes == 4) { // -> float
        float x, y, z;
        m_ifs->read(reinterpret_cast<char*>(&x), m_size_bytes);
        m_ifs->read(reinterpret_cast<char*>(&y), m_size_bytes);
        m_ifs->read(reinterpret_cast<char*>(&z), m_size_bytes);
        m_curr_point = Vector3(x, y, z);
      }else { // 8 bytes -> double
        double x, y, z;
        m_ifs->read(reinterpret_cast<char*>(&x), m_size_bytes);
        m_ifs->read(reinterpret_cast<char*>(&y), m_size_bytes);
        m_ifs->read(reinterpret_cast<char*>(&z), m_size_bytes);
        m_curr_point = Vector3(x, y, z);          
      }
    
    } else { // Text format

      // Read in the next point
      double x, y, z;
      (*m_ifs) >> x >> y >> z;
      m_curr_point = Vector3(x, y, z);
    }
    
    // Make sure the reads succeeded
    if (m_ifs->fail()) {
      m_has_valid_point = false;
      return false;
    }
    
    return true;
  }

  Vector3 PcdReader::GetPoint(){
    return m_curr_point;
  }

  PcdReader::~PcdReader(){
    delete m_ifs;
    m_ifs = NULL;
  }

  std::int64_t pcd_file_size(std::string const& file) {
    PcdReader reader(file);
    return reader.m_num_points;
  }

  /// Create a point cloud image from a las file. The image will be
  /// created block by block, when it needs to be written to disk. It is
  /// important that the writer invoking this image be single-threaded,
  /// as we read from the las file sequentially.
  template <class ImageT>
  class LasOrCsvToTif_Class : public ImageViewBase< LasOrCsvToTif_Class<ImageT> > {

    typedef typename ImageT::pixel_type PixelT;

    asp::BaseReader * m_reader;
    int m_rows, m_cols; // These are pixel sizes, not tile counts.
    int m_block_size;

  public:

    typedef PixelT pixel_type;
    typedef PixelT result_type;
    typedef ProceduralPixelAccessor<LasOrCsvToTif_Class> pixel_accessor;

    LasOrCsvToTif_Class(asp::BaseReader * reader, int num_rows, int tile_len, int block_size):
      m_reader(reader), m_block_size(block_size){

      std::int64_t num_points = m_reader->m_num_points;
      int num_row_tiles = std::max(1, (int)ceil(double(num_rows)/tile_len));
      m_rows = tile_len*num_row_tiles;

      int points_per_row = (int)ceil(double(num_points)/m_rows);
      int num_col_tiles  = std::max(1, (int)ceil(double(points_per_row)/tile_len));
      m_cols = tile_len*num_col_tiles;
    }

    inline int32 cols  () const { return m_cols; }
    inline int32 rows  () const { return m_rows; }
    inline int32 planes() const { return 1; }

    inline pixel_accessor origin() const { return pixel_accessor(*this); }

    inline result_type operator()( size_t i, size_t j, size_t p=0 ) const {
      vw_throw( NoImplErr() << "LasOrCsvToTif_Class::operator(...) has not been implemented.\n");
      return result_type();
    }

    typedef CropView<ImageView<PixelT> > prerasterize_type;
    inline prerasterize_type prerasterize( BBox2i const& bbox ) const{

      // Read a chunk of the las file, and store it in the current tile.

      std::int64_t num_cols = bbox.width();
      std::int64_t num_rows = bbox.height();

      VW_ASSERT((num_rows % m_block_size == 0) && (num_cols % m_block_size == 0),
                ArgumentErr() << "LasOrCsvToTif_Class: Expecting the number of rows "
                              << "to be a multiple of the block size.\n");

      // Read the specified number of points from the file
      std::int64_t max_num_pts_to_read = num_cols * num_rows;
      std::int64_t count = 0;
      PointBuffer in;
      while (m_reader->ReadNextPoint()){
        in.push_back(m_reader->GetPoint());
        count++;
        if (count >= max_num_pts_to_read)
          break;
      }

      // Take the points just read, and put them in groups by spatial
      // location, so that later point2dem does not need to read every
      // input point when writing a given tile, but only certain groups.
      ImageView<Vector3> Img;
      Chipper(in, m_block_size, m_reader->m_has_georef, m_reader->m_georef,
              num_cols, num_rows, Img);

      VW_ASSERT(num_cols == Img.cols() && num_rows == Img.rows(),
                ArgumentErr() << "LasOrCsvToTif_Class: Size mis-match.\n");

      return crop( Img, -bbox.min().x(), -bbox.min().y(), cols(), rows() );

    }

    template <class DestT>
    inline void rasterize( DestT const& dest, BBox2i const& bbox ) const {
      vw::rasterize( prerasterize(bbox), dest, bbox );
    }

  }; // End class LasOrCsvToTif_Class

} // namespace asp

//------------------------------------------------------------------------------------------
// Class CsvConv functions

std::string asp::CsvConv::write_header_string(std::string const delimiter) const{
  std::stringstream s;
  std::map<int, std::string>::const_iterator it;
  for (it = this->col2name.begin(); it != this->col2name.end(); it++)
    s << it->second << delimiter;
  return s.str();
}

// This is a complete list of all supported column names, it must be kept up to date.
int asp::CsvConv::get_sorted_index_for_name(std::string const& name){
  if (name == "file"     ) return 3; // The string goes in a different location
  if (name == "lon"      ) return 0;
  if (name == "lat"      ) return 1;
  if (name == "radius_m" ) return 2;
  if (name == "radius_km") return 2;
  if (name == "x"        ) return 0;
  if (name == "y"        ) return 1;
  if (name == "z"        ) return 2;
  if (name == "pixel_x"  ) return 0;
  if (name == "pixel_y"  ) return 1;
  if (name == "pixel_val") return 2;
  if (name == "easting"  ) return 0;
  if (name == "northing" ) return 1;
  if (name == "height_above_datum") return 2;

  vw_throw( ArgumentErr() << "Unsupported column name: " << name );
}

// Parse the CSV format string and build the data structure which
// will enable to convert from CSV to Cartesian and vice-versa.
// The user specifies 3 fields that determine the coordinate.
// If min_num_fields 2, autocomplete the third value to 0.
void asp::CsvConv::parse_csv_format(std::string const& csv_format_str,
                                    std::string const& csv_proj4_str,
                                    int min_num_fields) {

  // Make sure that these custom terms do not appear in the proj4 string.
  if ((csv_proj4_str.find("D_MOON") != std::string::npos) ||
      (csv_proj4_str.find("D_MARS") != std::string::npos)) {
    vw_throw(ArgumentErr() << "D_MOON and D_MARS are not official proj4 names."
                           << "Specify the datum elsewhere or define radii manually.\n");
  }

  *this = asp::CsvConv(); // Reset this object to the default state

  this->csv_format_str = csv_format_str; // Record inputs
  this->csv_proj4_str  = csv_proj4_str;

  std::string local = csv_format_str; // Make lowercase
  boost::algorithm::to_lower(local);

  if (local == "") // No format string passed in
    return;

  boost::replace_all(local, ":", " "); // Convert to space-delimited
  boost::replace_all(local, ",", " ");
  std::istringstream is(local);

  // The case of utm: "utm:23N 1:x 2:y 3:height_above_datum". Parse
  // the initial bit to get utm_zone and utm_north, leave the rest
  // alone.
  std::string str;
  is >> str;
  if (str == "utm"){
    is >> str;
    asp::parse_utm_str(str, this->utm_zone, this->utm_north);
  }else{
    // Go back to the original string
    is.clear();
    is.str(local);
  }

  int col;
  std::string name;
  while (is.good()) {
    // Grab the next two elements
    if (! (is >> col >> name))
      vw_throw(ArgumentErr() << "Could not parse: '" << csv_format_str << "'\n");

    // Convert to zero-based indexing and error check
    col--;
    if (col < 0 || this->col2name.count(col))
      vw_throw(ArgumentErr() << "Illegal column index in: '" << csv_format_str << "'\n");
    
    // Store in the lookup maps
    this->name2col[name] = col;
    this->col2name[col]  = name;
  }
  this->num_fields = this->name2col.size();
  const int MAX_NUM_FIELDS = 4; // Location and a file
  if ((this->num_fields < min_num_fields) || (this->num_fields > MAX_NUM_FIELDS))
    vw_throw(ArgumentErr() << "Invalid number of column indices in: '" << csv_format_str << "'\n");
  if (min_num_fields < 2) 
    vw::vw_throw(vw::ArgumentErr() << "Expecting at least two fields in the csv format.\n");
  
  // Sort the names into a pre-specified order.
  std::vector<std::string> sorted_names(this->num_fields);
  for (auto it = this->name2col.begin(); it != this->name2col.end(); it++){
    int index = get_sorted_index_for_name(it->first);
    sorted_names[index] = it->first;
    if (it->first != "file") // Only the point data goes into a vector, not the filename
      this->col2sort[it->second] = index;
  }

  // If only two fields are set, auto-complete the third. Values for that field
  // will be set to 0. This makes it convenient to use many of the functions
  // expecting 3 fields.
  if (this->num_fields == 2 && sorted_names.size() == 2) {
    if (sorted_names[0] == "x" && sorted_names[1] == "y")
      sorted_names.push_back("z");
    else if (sorted_names[0] == "lon" && sorted_names[1] == "lat")
      sorted_names.push_back("height_above_datum");
    else if (sorted_names[0] == "easting"  && sorted_names[1] == "northing")
      sorted_names.push_back("height_above_datum");
    else if (sorted_names[0] == "pixel_x" && sorted_names[1] == "pixel_y")
      sorted_names.push_back("pixel_val");
    else
      sorted_names.push_back("not_found"); // will trigger a failure below
  }
  
  // From the input strings, determine which set type applies to this file.
  if (sorted_names[0] == "x" &&
      sorted_names[1] == "y" &&
      sorted_names[2] == "z") {
    this->format = XYZ;
  }else if (sorted_names[0] == "lon" &&
            sorted_names[1] == "lat" &&
            sorted_names[2] == "radius_m"){
    this->format = LAT_LON_RADIUS_M;
  }else if (sorted_names[0] == "lon" &&
            sorted_names[1] == "lat" &&
            sorted_names[2] == "radius_km"){
    this->format = LAT_LON_RADIUS_KM;
  }else if (sorted_names[0] == "lon" &&
            sorted_names[1] == "lat" &&
            sorted_names[2] == "height_above_datum"){
    this->format = HEIGHT_LAT_LON;
  }else if (sorted_names[0] == "easting"  &&
            sorted_names[1] == "northing" &&
            sorted_names[2] == "height_above_datum"){
    this->format = EASTING_HEIGHT_NORTHING;
  } else if (sorted_names[0] == "pixel_x" &&
             sorted_names[1] == "pixel_y" &&
             sorted_names[2] == "pixel_val"){
    this->format = PIXEL_XYVAL;
  }else{
    vw_throw( ArgumentErr() << "Cannot understand the csv format string: "
                            << csv_format_str << ".\n" );
  }
}

bool asp::CsvConv::parse_georef(vw::cartography::GeoReference & georef) const {
  // If the user passed in a csv file containing easting, northing, height
  // above datum, and either a utm zone or a custom proj4 string,
  // pass that info into the georeference for the purpose of converting
  // later from easting and northing to lon and lat.

  if (this->utm_zone >= 0) { // UTM case
    try{
      georef.set_UTM(this->utm_zone, this->utm_north);
      return true;
    } catch ( const std::exception& e ) {
      vw_throw(ArgumentErr() << "Detected error: " << e.what()
                             << "\nPlease check if you are using an Earth datum.\n");
    }
  } else if (this->csv_proj4_str != "") { // Not UTM, with proj4 string
    bool have_user_datum = false, have_input_georef = false;
    Datum user_datum;
    asp::set_srs_string(this->csv_proj4_str, have_user_datum, user_datum,
                        have_input_georef, georef);
    return true;
  }else{ // No UTM, no proj4 string
    if (this->format == EASTING_HEIGHT_NORTHING)
      vw_throw( ArgumentErr() << "When a CSV file has easting and northing, the PROJ.4 string must be set via --csv-proj4.\n" );
  }
  return false;
}

asp::CsvConv::CsvRecord asp::CsvConv::parse_csv_line(bool & is_first_line, bool & success,
                                                     std::string const& line) const {
  // Parse a CSV file line in given format
  success = true;

  // Copy the input line into a temporary buffer
  const int bufSize = 2048;
  char temp[bufSize];
  strncpy(temp, line.c_str(), bufSize);

  std::string sep = asp::csv_separator();

  int col_index = -1; // The current column we are reading
  int num_floats_read = 0;
  int num_values_read = 0;

  CsvRecord values;

  // Quietly ignore empty lines, lines with spaces only, and lines starting with comments
  if (line.empty() || line[0] == '#' || hasSpacesOnly(line)) {
    success = false;
    is_first_line = false;
    return values;
  }
  
  char * ptr = temp;
  while (1) {

    col_index++; // Increment the column counter
    const char* token = strtok(ptr, sep.c_str());  // Split line on seperator char
    ptr = NULL; // After the first call, strtok expects a null pointer as input.
    if (token == NULL) break; // no more tokens
    if (num_values_read >= this->num_fields) break; // read enough values

    // Check if this is one of the columns we need to read
    if (this->col2name.find(col_index) == this->col2name.end())
      continue;

    if (this->col2name.at(col_index) == "file") // This is a string input
      values.file = token;
    else {
      // Parse the floating point value from the token
      double val;
      int flag = sscanf(token, "%lg", &val);
      if (flag == 0){ // Handle parsing failure
        success = false;
        break;
      }
      values.point_data[num_floats_read] = val;
      num_floats_read++;
    }
    num_values_read++;

  } // End loop through columns

  // Check if enough values were read and for NaN values
  if (num_values_read != this->num_fields || values.point_data != values.point_data)
    success = false;

  if (!success) {
    if (!is_first_line) {
      // Not the header
      vw_out () << "Failed to read line: " << line << "\n";
    }
  }

  is_first_line = false;
  return values;
}

// Search for "color = red" and find "red". Return false on failure.
// Can handle uppercase strings, also "color=red" and "color red".
bool parse_color(std::string const& line, std::string & color) {

  color = ""; // reset the output
  if (line.empty() || line[0] == '#') 
    return false;

  std::string line_lc = boost::to_lower_copy(line); // make lowercase

  size_t pos = line_lc.find("color");
  if (pos == std::string::npos) 
    return false;

  pos += 5; // go past the color

  // Skip past spaces and equal sign
  while (pos < line_lc.size() && (line_lc[pos] < 'a' || line_lc[pos] > 'z'))
    pos++;
  
  if (pos >= line_lc.size())
    return false; // no color was set

  line_lc = line_lc.substr(pos);

  // Return the first token. This is useful if there are spaces and other things
  // afterward.
  std::istringstream iss(line_lc);
  std::string token;
  if (iss >> token) {
    color = token;
    return true;
  }
  
  return false;
}

size_t asp::CsvConv::read_csv_file(std::string const & file_path,
				   std::list<CsvRecord> & output_list) const {

  // Clear output object
  output_list.clear();

  // Open input file
  std::ifstream file( file_path.c_str() );
  if (!file)
    vw_throw(vw::IOErr() << "Unable to open file \"" << file_path << "\"");

  // Read through all the lines of the input file, parse each line, and build the output list.
  bool success;
  bool first_line = true; // TODO(oalexan1): Wipe this variable.
  std::string line = "";
  
  while (std::getline(file, line, '\n')) {
    
    CsvRecord new_record = asp::CsvConv::parse_csv_line(first_line, success, line);
    if (success)
      output_list.push_back(new_record);
    
    first_line = false;
  }

  file.close();

  return output_list.size();
}

// Reads an entire CSV file having polygons. Individual
// polygons are separated by a newline or some other unexpected text.
size_t asp::CsvConv::read_poly_file(std::string    const & file_path,
                                    std::list<CsvRecord> & output_list,
                                    std::vector<int>         & contiguous_blocks,
                                    std::vector<std::string> & colors) const {

  // Clear output object
  output_list.clear();

  std::string color = "green"; // some default
  if (colors.size() > 0) {
    color = colors[0]; // use as default color what is passed from outside
    colors.clear(); // so we can keep on pushing for each polygon
  }
  
  contiguous_blocks.clear();  
  contiguous_blocks.push_back(0);
  
  // Open the input file
  std::ifstream file( file_path.c_str() );
  if (!file)
    vw_throw(vw::IOErr() << "Unable to open file \"" << file_path << "\"");

  // Read through all the lines of the input file, parse each line, and build
  // the output list. We ignore failed lines and lines having colors, and use
  // them as separators between polygons.
  bool success;
  bool first_line = true; // TODO(oalexan1): Rename this to: quiet_on_failure.
  std::string line = "";
  
  while (std::getline(file, line, '\n')) {
    std::string local_color;
    if (parse_color(line, local_color)) {
      color = local_color; 
      while (colors.size() < contiguous_blocks.size())
        colors.push_back(color); // catch up on colors
    }
    
    CsvRecord new_record = asp::CsvConv::parse_csv_line(first_line, success, line);
    first_line = true; // Because for now the api changes this var. To fix at some point.
    
    if (success) {
      output_list.push_back(new_record);
      // This is a bugfix. If we are just starting a block, and there were some
      // invalid lines before, and some were colors, let the color of this 
      // block be the last collected color
      if (contiguous_blocks.size() > 0 && 
        contiguous_blocks.back() == 0 && colors.size() == contiguous_blocks.size()) {
        colors.back() = color;
      }
      // add an element to the last block
      contiguous_blocks.back()++; 
    } else {
      if (contiguous_blocks.back() > 0) {
        contiguous_blocks.push_back(0); // Add a new block
        while (colors.size() < contiguous_blocks.size())
          colors.push_back(color); // catch up on colors
      }
    }
  }

  // This is needed in case no colors were found
  while (colors.size() < contiguous_blocks.size())
    colors.push_back(color);

  // Wipe all blocks of length 0. Likely there is only one at the end.
  std::vector<int> & v = contiguous_blocks; // alias
  v.erase(std::remove(v.begin(), v.end(), 0), v.end());
  if (colors.size() > contiguous_blocks.size())
    colors.resize(contiguous_blocks.size()); 
  
  file.close();

  return output_list.size();
}

vw::Vector3 asp::CsvConv::sort_parsed_vector3(CsvRecord const& csv) const {
  Vector3 ordered_csv;
  int count = 0;
  const int NUM_POINT_PARAMS = 3;
  for (auto it = this->col2sort.begin(); it != this->col2sort.end(); it++){
    if (it->second < NUM_POINT_PARAMS) // Don't include elements past the first three
      ordered_csv[it->second] = csv.point_data[count];
    count++;
  }
  return ordered_csv;
}

vw::Vector3 asp::CsvConv::unsort_vector3(vw::Vector3 const& csv) const {
  Vector3 csv2;
  int count = 0;
  const int NUM_POINT_PARAMS = 3;
  for (std::map<int, int>::const_iterator it = this->col2sort.begin(); it != this->col2sort.end(); it++){
    if (it->second < NUM_POINT_PARAMS){ // Don't include elements past the first three
      csv2[count] = csv[it->second];
      count++;
    }
  }
  return csv2;
}

// There is a lot of repeated code for the next three functions in order to
// improve the speed of parsing points by doing the minimum number of conversions.

// Return either xyz or a projected point. Note that the flag return_point_height
// is not necessarily respected. 
Vector3 asp::CsvConv::csv_to_cartesian_or_point_height(CsvRecord const& csv,
                                                       GeoReference const& geo,
                                                       bool return_point_height) const{

  Vector3 ordered_csv = sort_parsed_vector3(csv);

  // If the format is XYZ, there is a good chance we don't even have a reference.
  // So we cannot return a point_height. We need things this way for the chipper,
  // but this is quite confusing. 
  if (this->format == XYZ) 
    return ordered_csv; // already as xyz

  // Convert from CSV to Cartesian. Later we may convert to point_height format,
  // which, due to the projection in the georeference, may not be the same
  // as the input CSV format. E.g., input CSV may be lon, lat, height,
  // but the output of this function may be easting, northing, height.
  
  Vector3 xyz;
  if (this->format == EASTING_HEIGHT_NORTHING){

  Vector3 point_height = Vector3(ordered_csv[0], ordered_csv[1], ordered_csv[2]);
  Vector2 ll  = geo.point_to_lonlat(Vector2(point_height[0], point_height[1]));
  Vector3 llh = Vector3(ll[0], ll[1], point_height[2]); // now lon, lat, height
  xyz = geo.datum().geodetic_to_cartesian(llh);
  
  }else if (this->format == HEIGHT_LAT_LON){

    xyz = geo.datum().geodetic_to_cartesian(ordered_csv);

  }else{ // Handle asp::LAT_LON_RADIUS_M and asp::LAT_LON_RADIUS_KM

    if (this->format == LAT_LON_RADIUS_KM)
      ordered_csv[2] *= 1000.0; // now lon, lat, radius_m

    Vector3 tmp = ordered_csv; tmp[2] = 0; // now lon, lat, 0
    xyz = geo.datum().geodetic_to_cartesian(tmp);

    // Update the radius
    xyz = ordered_csv[2]*(xyz/norm_2(xyz));

  }
  
  if (return_point_height)
    return geo.geodetic_to_point(geo.datum().cartesian_to_geodetic(xyz));
  
  return xyz;
}


vw::Vector3 asp::CsvConv::csv_to_cartesian(CsvRecord const& csv,
                                           vw::cartography::GeoReference const& geo) const {
  Vector3 ordered_csv = sort_parsed_vector3(csv);

  Vector3 xyz;
  if (this->format == XYZ){
    return ordered_csv; // already as xyz

  }else if (this->format == EASTING_HEIGHT_NORTHING){
    Vector3 point_height = Vector3(ordered_csv[0], ordered_csv[1], ordered_csv[2]);
    Vector2 ll           = geo.point_to_lonlat(Vector2(point_height[0], point_height[1]));
    Vector3 llh          = Vector3(ll[0], ll[1], point_height[2]); // now lon, lat, height
    xyz = geo.datum().geodetic_to_cartesian(llh);

  }else if (this->format == HEIGHT_LAT_LON){
    xyz = geo.datum().geodetic_to_cartesian(ordered_csv);

  }else{ // Handle asp::LAT_LON_RADIUS_M and asp::LAT_LON_RADIUS_KM
    if (this->format == LAT_LON_RADIUS_KM)
      ordered_csv[2] *= 1000.0; // now lon, lat, radius_m

    Vector3 tmp = ordered_csv; tmp[2] = 0; // now lon, lat, 0
    xyz = geo.datum().geodetic_to_cartesian(tmp);

    // Update the radius
    xyz = ordered_csv[2]*(xyz/norm_2(xyz));
  }
  return xyz;
}

// Returns Vector3(lon, lat, height_above_datum)
vw::Vector3 asp::CsvConv::csv_to_geodetic(CsvRecord const& csv,
                                          vw::cartography::GeoReference const& geo) const {
  Vector3 ordered_csv = sort_parsed_vector3(csv);
  Vector3 llh;

  if (this->format == XYZ){
    llh = geo.datum().cartesian_to_geodetic(ordered_csv);

  }else if (this->format == EASTING_HEIGHT_NORTHING){
    Vector3 point_height = Vector3(ordered_csv[0], ordered_csv[1], ordered_csv[2]);
    Vector2 ll           = geo.point_to_lonlat(Vector2(point_height[0], point_height[1]));
    llh = Vector3(ll[0], ll[1], point_height[2]); // now lon, lat, height

  }else if (this->format == HEIGHT_LAT_LON){
    return ordered_csv;

  }else{ // Handle asp::LAT_LON_RADIUS_M and asp::LAT_LON_RADIUS_KM
    if (this->format == LAT_LON_RADIUS_KM)
      ordered_csv[2] *= 1000.0; // now lon, lat, radius_m

    Vector3 tmp = ordered_csv; tmp[2] = 0; // now lon, lat, 0
    Vector3 xyz = geo.datum().geodetic_to_cartesian(tmp);

    // Update the radius
    xyz = ordered_csv[2]*(xyz/norm_2(xyz));
    llh = geo.datum().cartesian_to_geodetic(xyz);
  }
  return llh;
}

vw::Vector2 asp::CsvConv::csv_to_lonlat(CsvRecord const& csv,
                                        vw::cartography::GeoReference const& geo) const {
  Vector3 ordered_csv = sort_parsed_vector3(csv);

  if (this->format == XYZ){
    Vector3 llh = geo.datum().cartesian_to_geodetic(ordered_csv);
    return Vector2(llh[0], llh[1]);
  }else if (this->format == EASTING_HEIGHT_NORTHING){
    return geo.point_to_lonlat(Vector2(ordered_csv[0], ordered_csv[1]));
  }else if (this->format == HEIGHT_LAT_LON){
    return Vector2(ordered_csv[0], ordered_csv[1]);
  }else{ // Handle asp::LAT_LON_RADIUS_M and asp::LAT_LON_RADIUS_KM
    return Vector2(ordered_csv[0], ordered_csv[1]);
  }

}

Vector3 asp::CsvConv::cartesian_to_csv(Vector3 const& xyz,
                                       GeoReference const& geo,
                                       double mean_longitude) const{
  Vector3 csv;
  if (this->format == XYZ){
    csv = xyz; // order is x, y, z

  }else{ // format != XYZ, convert to the csv format.

    // Must assert here that the datum was specified.

    Vector3 llh = geo.datum().cartesian_to_geodetic(xyz);   // lon-lat-height
    llh[0] += 360.0*round((mean_longitude - llh[0])/360.0); // 360 deg adjust

    if (this->format == EASTING_HEIGHT_NORTHING){

      // go from lon, lat to easting, northing
      Vector2 en = geo.lonlat_to_point(Vector2(llh[0], llh[1]));
      csv = Vector3(en[0], en[1], llh[2]); // order is easting, northing, height

    }else if (this->format == HEIGHT_LAT_LON){
      csv = llh;

    }else{
      // Handle asp::LAT_LON_RADIUS_M and asp::LAT_LON_RADIUS_KM

      llh[2] = norm_2(xyz); // order is lon, lat, radius_m

      if (this->format == LAT_LON_RADIUS_KM){
        llh[2] /= 1000.0; // order is lon, lat, radius_km
      }
      csv = llh;
    }
  }
  // Now the csv vector contains the sorted values for our format

  // Now we have the csv fields, but they are in the order
  // corresponding to the sorted column names. Need to put them
  // in the same order as they were in the file originally.
  return unsort_vector3(csv);
}

// End class CsvConv functions

void asp::las_or_csv_to_tif(std::string const& in_file,
                            std::string const& out_file,
                            int num_rows, int block_size,
                            vw::GdalWriteOptions * opt,
                            vw::cartography::GeoReference const& csv_georef,
                            asp::CsvConv const& csv_conv) {

  // We will fetch a chunk of the las file of area TILE_LEN x
  // TILE_LEN, split it into bins of spatially close points, and write
  // it to disk as a tile in a vector tif image. The bigger the tile
  // size, the more likely the binning will be more efficient. But big
  // tiles use a lot of memory.

  // To do: Study performance for large files when this number changes
  const int TILE_LEN = 2048;
  Vector2 tile_size(TILE_LEN, TILE_LEN);

  vw_out() << "Writing temporary file: " << out_file << std::endl;

  // Temporarily change the raster tile size
  Vector2 original_tile_size = opt->raster_tile_size;
  opt->raster_tile_size = tile_size;

  boost::shared_ptr<asp::BaseReader> reader_ptr;
  std::ifstream ifs;
  liblas::ReaderFactory las_reader_factory;
  boost::shared_ptr<liblas::Reader> laslib_reader_ptr;

  if (asp::is_csv(in_file)){ // CSV

    reader_ptr = boost::shared_ptr<asp::CsvReader>
      (new asp::CsvReader(in_file, csv_conv, csv_georef));

  }else if (asp::is_pcd(in_file)){ // PCD

    reader_ptr = boost::shared_ptr<asp::PcdReader>( new asp::PcdReader(in_file) );

  }else if (asp::is_las(in_file)){ // LAS

    ifs.open(in_file.c_str(), std::ios::in | std::ios::binary);
    laslib_reader_ptr.reset(new liblas::Reader(las_reader_factory.CreateWithStream(ifs)));
    reader_ptr = boost::shared_ptr<asp::LasReader>(new asp::LasReader(*laslib_reader_ptr));

  }else
    vw_throw( ArgumentErr() << "Unknown file type: " << in_file << "\n");

  ImageViewRef<Vector3> Img
    = asp::LasOrCsvToTif_Class<ImageView<Vector3>>(reader_ptr.get(), num_rows,
                                                     TILE_LEN, block_size);

  // Must use a thread only, as we read the input file serially.
  vw::cartography::write_gdal_image(out_file, Img, *opt, TerminalProgressCallback("asp", "\t--> ") );

  // Restore the original tile size
  opt->raster_tile_size = original_tile_size;
}

std::int64_t asp::las_file_size(std::string const& las_file){
  std::ifstream ifs;
  ifs.open(las_file.c_str(), std::ios::in | std::ios::binary);
  liblas::ReaderFactory f;
  liblas::Reader reader = f.CreateWithStream(ifs);
  liblas::Header const& header = reader.GetHeader();
  return header.GetPointRecordsCount();
}

bool asp::georef_from_las(std::string const& las_file,
                          vw::cartography::GeoReference & georef){

  if (!is_las(las_file))
    vw_throw( ArgumentErr() << "Not a LAS file: " << las_file << "\n");

  // Initialize
  georef = GeoReference();

  std::ifstream ifs;
  ifs.open(las_file.c_str(), std::ios::in | std::ios::binary);
  liblas::ReaderFactory f;
  liblas::Reader reader = f.CreateWithStream(ifs);
  liblas::Header const& header = reader.GetHeader();

  std::string wkt = header.GetSRS().GetWKT();
  if (wkt == "")
    return false;

  georef.set_wkt(wkt);
  return true;
}

/// Builds a GeoReference from the first cloud having a georeference in the list
bool asp::georef_from_pc_files(std::vector<std::string> const& files,
			       vw::cartography::GeoReference & georef){

  // Initialize
  georef = GeoReference();

  for (int i = 0; i < (int)files.size(); i++){
    GeoReference local_georef;

    // Sometimes ASP PC files can have georef, written there by stereo
    try {
      if (!is_las(files[i]) && read_georeference(local_georef, files[i])){
	georef = local_georef;
	return true;
      }
    }catch(...){}

    // Sometimes las files can have georef
    if (is_las(files[i]) && asp::georef_from_las(files[i], local_georef)){
      georef = local_georef;
      return true;
    }
  }

  return false;
}

bool asp::is_las(std::string const& file){
  std::string lfile = boost::to_lower_copy(file);
  return (boost::iends_with(lfile, ".las")  || boost::iends_with(lfile, ".laz"));
}

bool asp::is_csv(std::string const& file){
  std::string lfile = boost::to_lower_copy(file);
  return ( boost::iends_with(lfile, ".csv")  || boost::iends_with(lfile, ".txt")  );
}

bool asp::is_pcd(std::string const& file){
  std::string lfile = boost::to_lower_copy(file);
  return boost::iends_with(lfile, ".pcd");
}

bool asp::is_las_or_csv_or_pcd(std::string const& file){
  return asp::is_las(file) || is_csv(file);
}

bool asp::read_user_datum(double semi_major, double semi_minor,
                          std::string const& reference_spheroid,
                          cartography::Datum& datum ) {
  // Select a cartographic datum. There are several hard coded datums
  // that can be used here, or the user can specify their own.
  if ( reference_spheroid != "" ) {
    datum.set_well_known_datum(reference_spheroid);
  } else if (semi_major > 0 && semi_minor > 0) {
    datum = cartography::Datum("User Specified Datum",
                               "User Specified Spheroid",
                               "Reference Meridian",
                               semi_major, semi_minor, 0.0);
  } else {
    return false;
  }
  vw_out() << "\t--> Re-referencing altitude values using datum: " << datum.name() << ".\n";
  vw_out() << "\t    Axes [" << datum.semi_major_axis() << " "
           << datum.semi_minor_axis() << "] meters.\n";
  return true;
}


void asp::parse_utm_str(std::string const& utm, int & zone, bool & north){

  // Parse the string 58N

  // Initialize
  zone = -1; north = false;

  std::string a, b;
  for (int s = 0; s < (int)utm.size(); s++){
    if (utm[s] >= '0' && utm[s] <= '9'){
      a += utm[s];
    }else{
      b = utm[s];
      break;
    }
  }

  if (a == "" || b == "")
    vw_throw(ArgumentErr() << "Could not parse UTM string: '" << utm << "'\n");

  zone = atoi(a.c_str());
  if (b == "n" || b == "N"){
    north = true;
  }else if (b == "s" || b == "S"){
    north = false;
  }else
    vw_throw(ArgumentErr() << "Could not parse UTM string: '" << utm << "'\n");
}

bool asp::hasSpacesOnly(std::string const& str) {
  bool only_spaces = true;
  for (size_t it = 0; it < str.size(); it++) {
    if (str[it] != ' ' && str[it] != '\n' && str[it] != '\t') {
      only_spaces = false;
      break;
    }
  }
  return only_spaces;
}

bool asp::is_valid_csv_line(std::string const& line) {
  // A valid line is not empty and does not start with '#' and does not have spaces only.

  bool only_spaces = hasSpacesOnly(line);
  
  return (!only_spaces) && (!line.empty()) && (line[0] != '#');
}

std::int64_t asp::csv_file_size(std::string const& file){

  std::ifstream fh(file.c_str());
  if (!fh)
    vw_throw( vw::IOErr() << "Unable to open file \"" << file << "\"" );

  std::int64_t num_total_points = 0;
  std::string line;
  while (getline(fh, line, '\n')){
    if (!asp::is_valid_csv_line(line)) continue;
    num_total_points++;
  }

  return num_total_points;
}

// Peek at the first valid line in a file to find how many columns it has
int asp::fileNumCols(std::string const& file) {

  const int bufSize = 2048;
  char buffer[bufSize];

  std::string sep = asp::csv_separator();
  
  int num = 0;
  std::ifstream fh(file.c_str());

  std::string line;
  while (getline(fh, line, '\n')){
    if (!asp::is_valid_csv_line(line)) continue;

    // Copy the input line into a buffer that can be modified
    strncpy(buffer, line.c_str(), bufSize);
    char * ptr = buffer;

    // Inspect the tokens
    while (1) {
      
      const char* token = strtok(ptr, sep.c_str());  // Split line on seperator char
      ptr = NULL; // After the first call, strtok expects a null pointer as input.

      if (token == NULL)
        break; // no more tokens
      
      // Parse the floating point value from the token
      double val;
      int flag = sscanf(token, "%lg", &val);
      if (flag == 0) // Handle parsing failure
        break;

      num++;
    }

    break; // done finding a good line
  }
  
  return num;
}

// Erases a file suffix if one exists and returns the base string
std::string asp::prefix_from_pointcloud_filename(std::string const& filename) {
  std::string result = filename;

  // First case: filenames that match <prefix>-PC.<suffix>
  int index = result.rfind("-PC.");
  if (index != -1) {
    result.erase(index, result.size());
    return result;
  }

  // Second case: filenames that match <prefix>.<suffix>
  index = result.rfind(".");
  if (index != -1) {
    result.erase(index, result.size());
    return result;
  }

  // No match
  return result;
}

// Compute bounding box of the given cloud. If is_geodetic is false,
// that means a cloud of raw xyz cartesian values, then Vector3()
// signifies no-data. If is_geodetic is true, no-data is suggested
// by having the z component of the point be NaN.
vw::BBox3 asp::pointcloud_bbox(vw::ImageViewRef<vw::Vector3> const& point_image,
                               bool is_geodetic) {

  vw::BBox3 result;
  vw::vw_out() << "Computing the point cloud bounding box.\n";
  vw::TerminalProgressCallback progress_bar("asp", "\t--> ");

  for (int row=0; row < point_image.rows(); ++row ) {
    progress_bar.report_fractional_progress(row, point_image.rows());
    for (int col=0; col < point_image.cols(); ++col ) {
      vw::Vector3 pt = point_image(col, row);
      if ( (!is_geodetic && pt != vw::Vector3()) ||
           (is_geodetic  &&  !boost::math::isnan(pt.z())) )
        result.grow(pt);
    }
  }
  progress_bar.report_finished();

  return result;
}

// Find the average longitude for a given point image with lon, lat, height values
double asp::find_avg_lon(ImageViewRef<Vector3> const& point_image){

  Stopwatch sw;
  sw.start();
  int32 subsample_amt = int32(norm_2(Vector2(point_image.cols(),
                                             point_image.rows()))/32.0);
  if (subsample_amt < 1 )
    subsample_amt = 1;
  PixelAccumulator<MeanAccumulator<Vector3> > mean_accum;
  for_each_pixel( subsample(point_image, subsample_amt),
                  mean_accum,
                  TerminalProgressCallback("asp","Statistics: ") );
  Vector3 avg_location = mean_accum.value();
  double avg_lon = avg_location.x() >= 0 ? 0 : 180;
  sw.stop();
  vw_out(DebugMessage,"asp") << "Statistics time: " << sw.elapsed_seconds() << std::endl;

  return avg_lon;
}


/// Analyze a file name to determine the file type
std::string asp::get_cloud_type(std::string const& file_name){

  if (asp::is_csv(file_name))
    return "CSV";
  if (asp::is_las(file_name))
    return "LAS";

  // Note that any tif, ntf, and cub file with one channel with georeference be
  // interpreted as a DEM.
  int nc = vw::get_num_channels(file_name);

  vw::cartography::GeoReference geo;
  bool has_georef = vw::cartography::read_georeference(geo, file_name);

  if (nc == 1 && has_georef)
    return "DEM";
  if (nc >= 3)
    return "PC";
  vw_throw(vw::ArgumentErr() << "File: " << file_name
                         << " is neither a point cloud nor a DEM.\n");
}

// Find the number of channels in the point clouds.
// If the point clouds have inconsistent number of channels,
// return the minimum of 3 and the minimum number of channels.
// This will be used to flag that we cannot reliable extract the
// error channels, which start at channel 4.
int asp::num_channels(std::vector<std::string> const& pc_files) {

  VW_ASSERT(pc_files.size() >= 1, ArgumentErr() << "Expecting at least one point cloud file.\n");

  int num_channels0 = get_num_channels(pc_files[0]);
  int min_num_channels = num_channels0;
  for (int i = 1; i < (int)pc_files.size(); i++){
    int num_channels = get_num_channels(pc_files[i]);
    min_num_channels = std::min(min_num_channels, num_channels);
    if (num_channels != num_channels0)
      min_num_channels = std::min(min_num_channels, 3);
  }
  return min_num_channels;
}

// See if all the input point cloud files have stddev values
bool asp::has_stddev(std::vector<std::string> const& pc_files) {
  
  VW_ASSERT(pc_files.size() >= 1, ArgumentErr() << "Expecting at least one point cloud file.\n");

  bool has_sd = true;
  for (size_t i = 0; i < pc_files.size(); i++) {
    std::string val;
    std::string adj_key = "BAND5";
    boost::shared_ptr<vw::DiskImageResource> rsrc(new vw::DiskImageResourceGDAL(pc_files[i]));
    vw::cartography::read_header_string(*rsrc.get(), adj_key, val);
    if (val != "HorizontalStdDev")
      has_sd = false;
  }

  if (has_sd && asp::num_channels(pc_files) < 6) 
    has_sd = false;

  return has_sd;
}

// Get a handle to the error image given a set of point clouds with 4 or 6 bands
vw::ImageViewRef<double> asp::point_cloud_error_image
(std::vector<std::string> const& pointcloud_files) {

  ImageViewRef<double> error_image;
  int num_channels = asp::num_channels(pointcloud_files);
  bool has_sd = asp::has_stddev(pointcloud_files);
  
  if (num_channels == 4 || (num_channels == 6 && has_sd)) {
    // The error is a scalar (4 channels or 6 channels but last two are stddev values)
    error_image = asp::error_norm<4>(pointcloud_files);
  } else if (num_channels == 6) {
    error_image = asp::error_norm<6>(pointcloud_files);
  } else {
    // Return an empty image
    ImageView<double> image;
    image.set_size(0, 0);
    error_image = image;
  }

  return error_image;
}
