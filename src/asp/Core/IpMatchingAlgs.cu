#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Core/IpMatchingAlgs.h>         // Lightweight header
#include <vw/InterestPoint/InterestData.h>
#include <vw/InterestPoint/Matcher.h>
#include <vw/Camera/CameraModel.h>
#include <boost/filesystem.hpp>
using namespace vw;
namespace fs = boost::filesystem;

namespace asp {

// Outlier removal based on the disparity of interest points.
// Points with x or y disparity not within the 100-'pct' to 'pct'
// percentile interval expanded by 'factor' will be removed as
// outliers. Overwrite the ip in place.
void filter_ip_by_disparity(double pct,    // for example, 90.0
                            double factor, // for example, 3.0
                            bool quiet,    // if not to print a lot of text
                            std::vector<vw::ip::InterestPoint> & left_ip,
                            std::vector<vw::ip::InterestPoint> & right_ip){

  double pct_fraction = 1.0 - pct/100.0;
  double bx, ex, by, ey;
  std::vector<double> dispx, dispy;
  for (size_t it = 0; it < left_ip.size(); it++) {
    dispx.push_back(right_ip[it].x - left_ip[it].x);
    dispy.push_back(right_ip[it].y - left_ip[it].y);
  }
  vw::math::find_outlier_brackets(dispx, pct_fraction, factor, bx, ex);
  vw::math::find_outlier_brackets(dispy, pct_fraction, factor, by, ey);
    
  //vw_out() << "Outlier statistics by disparity in x: b = " << bx << ", e = " << ex << ".\n";
  //vw_out() << "Outlier statistics by disparity in y: b = " << by << ", e = " << ey << ".\n";
    
  // Remove the bad ip 
  size_t good_it = 0;
  for (size_t it = 0; it < left_ip.size(); it++) {
    if (dispx[it] < bx || dispx[it] > ex) continue;
    if (dispy[it] < by || dispy[it] > ey) continue;
    left_ip [good_it] = left_ip[it];
    right_ip[good_it] = right_ip[it];
    good_it++;
  }

  if (!quiet)
    vw_out() << "Removed " << left_ip.size() - good_it
             << " outliers based on percentiles of differences of interest "
             << "points with --outlier-removal-params.\n";
  
  left_ip.resize(good_it);
  right_ip.resize(good_it);

  return;
}


double calc_ip_coverage_fraction(std::vector<ip::InterestPoint> const& ip,
                                 vw::Vector2i const& image_size, int tile_size,
                                 int min_ip_per_tile) {

  if (tile_size < 1)
    vw_throw(LogicErr() << "calc_ip_coverage_fraction: tile size is " << tile_size);

  // Generate a grid of ROIs covering the entire image
  BBox2i full_bbox(Vector2i(0,0), image_size);
  bool include_partials = false;
  std::vector<BBox2i> rois;
  rois = subdivide_bbox(full_bbox, tile_size, tile_size, include_partials);
  const size_t num_rois = rois.size();
  if (num_rois == 0)
    return 0; // Cannot have any coverage in the degenerate case!
    
  // Pack all IP into a list for speed
  std::list<Vector2i> ip_list;
  for (size_t i=0; i<ip.size(); ++i) {
    ip_list.push_back(Vector2i(ip[i].x, ip[i].y));
  }
    
  size_t num_filled_rois = 0;
  for (size_t i=0; i<num_rois; ++i) { // Loop through ROIs
    int ip_in_roi = 0;
      
    // Check if each point is in this ROI
    std::list<Vector2i>::iterator iter;
    for (iter=ip_list.begin(); iter!=ip_list.end(); ++iter) {
        
      // If the IP is in the ROI, remove it from the IP list so it
      // does not get searched again.
      if (rois[i].contains(*iter)) {
        iter = ip_list.erase(iter);
        ++ip_in_roi;
        --iter;
      }
    } // End IP loop
    if (ip_in_roi > min_ip_per_tile)
      ++num_filled_rois;
  }// End ROI loop

  return static_cast<double>(num_filled_rois) / static_cast<double>(num_rois);
}
  
/// Apply alignment transform to ip. Not to be used with mapprojected images.
void align_ip(vw::TransformPtr const& tx_left,
              vw::TransformPtr const& tx_right,
              std::vector<vw::ip::InterestPoint> & ip_left,
              std::vector<vw::ip::InterestPoint> & ip_right) {

  // Loop through all the IP we found
  for (size_t i = 0; i < ip_left.size(); i++) {
    // Apply the alignment transforms to the recorded IP
    Vector2 l = tx_left->forward (Vector2(ip_left [i].x,  ip_left [i].y));
    Vector2 r = tx_right->forward(Vector2(ip_right[i].x,  ip_right[i].y));

    ip_left [i].x = l[0];
    ip_left [i].y = l[1];
    ip_left [i].ix = l[0];
    ip_left [i].iy = l[1];
    
    ip_right[i].x = r[0];
    ip_right[i].y = r[1];
    ip_right[i].ix = r[0];
    ip_right[i].iy = r[1];
  }

  return;
} // End align_ip

// Heuristics for match file prefix
std::string match_file_prefix(std::string const& clean_match_files_prefix,
                              std::string const& match_files_prefix,
                              std::string const& out_prefix) {
  
  if (clean_match_files_prefix != "")
    return clean_match_files_prefix;
  else if (match_files_prefix != "")
    return match_files_prefix;
  return out_prefix; 
}
  
// Heuristics for where to load the match file from  
std::string match_filename(std::string const& clean_match_files_prefix,
                           std::string const& match_files_prefix,
                           std::string const& out_prefix,
                           std::string const& image1_path,
                           std::string const& image2_path) {

  std::string curr_prefix = asp::match_file_prefix(clean_match_files_prefix,
                                              match_files_prefix,  
                                              out_prefix);

  if (clean_match_files_prefix != "")
    return vw::ip::clean_match_filename(curr_prefix, image1_path, image2_path);

  return vw::ip::match_filename(curr_prefix, image1_path, image2_path);
}

/// The unwarped disparity file name
std::string unwarped_disp_file(std::string const& prefix, std::string const& left_image,
                               std::string const& right_image){
    
  std::string match_file = vw::ip::match_filename(prefix, left_image, right_image);

  std::string disp_file = boost::filesystem::path(match_file).replace_extension("").string();
  return disp_file + "-unaligned-D.tif";
}
  
// Find and sort the convergence angles for given cameras and interest points
void convergence_angles(vw::camera::CameraModel const * left_cam,
                        vw::camera::CameraModel const * right_cam,
                        std::vector<vw::ip::InterestPoint> const& left_ip,
                        std::vector<vw::ip::InterestPoint> const& right_ip,
                        std::vector<double> & sorted_angles) {

  int num_ip = left_ip.size();
  sorted_angles.clear();
  for (int ip_it = 0; ip_it < num_ip; ip_it++) {
    Vector2 lip(left_ip[ip_it].x,  left_ip[ip_it].y);
    Vector2 rip(right_ip[ip_it].x, right_ip[ip_it].y);
    double angle = 0.0;
    try {
      angle = (180.0 / M_PI) * acos(dot_prod(left_cam->pixel_to_vector(lip),
                                             right_cam->pixel_to_vector(rip)));
    } catch(...) {
      // Projection into camera may not always succeed
      continue;
    }
      
    if (std::isnan(angle)) 
      continue;
      
    sorted_angles.push_back(angle);
  }
  
  std::sort(sorted_angles.begin(), sorted_angles.end());
}

// Find all match files stored on disk having this prefix
void listExistingMatchFiles(std::string const& prefix,
                            std::set<std::string> & existing_files) {
  existing_files.clear();
  
  fs::path dirName = fs::path(prefix).parent_path().string();
  for (auto i = fs::directory_iterator(dirName); i != fs::directory_iterator(); i++) {
    if (fs::is_directory(i->path())) // skip dirs
      continue;
    std::string filename = i->path().string();
    if (filename.find(".match") != std::string::npos)
      existing_files.insert(filename);
  }
}
  
} // end namespace asp
