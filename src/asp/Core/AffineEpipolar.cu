#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Core/AffineEpipolar.h>
#include <asp/Core/OpenCVUtils.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Core/InterestPointMatching.h>  // Slow-to-compile header
#include <asp/Core/IpMatchingAlgs.h>         // Lightweight header
#include <vw/Math/Vector.h>
#include <vw/Math/Matrix.h>
#include <vw/Math/RANSAC.h>
#include <vw/Math/LinearAlgebra.h>
#include <vw/InterestPoint/InterestData.h>
#include <vw/Core/Stopwatch.h>
#include <vw/Math/Transform.h>

#include <opencv2/calib3d.hpp>

#include <vector>

using namespace vw;
using namespace vw::math;

namespace asp {

  // Solves for Affine Fundamental Matrix as per instructions in
  // Multiple View Geometry. Outlier elimination happens later. 
  Matrix<double>
  linear_affine_fundamental_matrix(std::vector<ip::InterestPoint> const& ip1,
                                   std::vector<ip::InterestPoint> const& ip2) {

    // (i) Compute the centroid of X and delta X
    Matrix<double> delta_x(ip1.size(), 4);
    Vector4 mean_x;
    for (size_t i = 0; i < ip1.size(); i++) {
      delta_x(i, 0) = ip2[i].x;
      delta_x(i, 1) = ip2[i].y;
      delta_x(i, 2) = ip1[i].x;
      delta_x(i, 3) = ip1[i].y;
      mean_x += select_row(delta_x, i) / double(ip1.size());
    }
    
    for (size_t i = 0; i < ip1.size(); i++) 
      select_row(delta_x,i) -= mean_x;

    Matrix<double> U, VT;
    Vector<double> S;
    svd(transpose(delta_x), U, S, VT);
    Vector<double> N = select_col(U, 3);
    double e = -transpose(N) * mean_x;
    Matrix<double> f(3,3);
    f(0,2) = N(0);
    f(1,2) = N(1);
    f(2,2) = e;
    f(2,0) = N(2);
    f(2,1) = N(3);

    return f;
  }

  void solve_y_scaling(std::vector<ip::InterestPoint> const & ip1,
                       std::vector<ip::InterestPoint> const & ip2,
                       Matrix<double>                       & affine_left,
                       Matrix<double>                       & affine_right) {
    
    Matrix<double> a(ip1.size(), 2);
    Vector<double> b(ip1.size());
    
    for (size_t i = 0; i < ip1.size(); i++) {
      select_row(a, i) = subvector(affine_right*Vector3(ip2[i].x, ip2[i].y, 1), 1, 2);
      b[i]             = (affine_left*Vector3(ip1[i].x, ip1[i].y, 1))(1);
    }

    Vector<double> scaling = least_squares(a, b);
    submatrix(affine_right,0,0,2,2) *= scaling[0];
    affine_right(1,2) = scaling[1];
  }
  
  void solve_x_shear(std::vector<ip::InterestPoint> const & ip1,
                     std::vector<ip::InterestPoint> const & ip2,
                     Matrix<double>                       & affine_left,
                     Matrix<double>                       & affine_right) {
    
    Matrix<double> a(ip1.size(), 3);
    Vector<double> b(ip1.size());
    
    for (size_t i = 0; i < ip1.size(); i++) {
      select_row(a, i) = affine_right * Vector3(ip2[i].x, ip2[i].y, 1);
      b[i] = (affine_left * Vector3(ip1[i].x, ip1[i].y, 1))(0);
    }

    Vector<double> shear = least_squares(a, b);
    Matrix<double> interm = math::identity_matrix<3>();
    interm(0, 1) = -shear[1] / 2.0;
    affine_left = interm * affine_left;
    interm = math::identity_matrix<3>();
    interm(0, 0) = shear[0];
    interm(0, 1) = shear[1] / 2.0;
    interm(0, 2) = shear[2];
    affine_right = interm * affine_right;
  }

  // A functor which returns the best fit left and right 3x3 matrices
  // for epipolar alignment. Store them as a single 3x7 matrix.  The
  // last column will have the upper-right corner of the intersections
  // of the domains of the left and right images with the resulting
  // transformed applied to them.

  struct BestFitEpipolarAlignment {

    Vector2i m_ldims, m_rdims;
    bool m_crop_to_shared_area;
    
    BestFitEpipolarAlignment(Vector2i const& left_image_dims,
                             Vector2i const& right_image_dims,
                             bool crop_to_shared_area):
      m_ldims(left_image_dims), m_rdims(right_image_dims),
      m_crop_to_shared_area(crop_to_shared_area) {}

    typedef vw::Matrix<double, 3, 7> result_type;

    /// The fundamental matrix needs 8 points.
    // TODO(oalexan1): Should a bigger minimum be used for robustness?
    template <class InterestPointT>
    size_t min_elements_needed_for_fit(InterestPointT const& /*example*/) const {
      return 8;
    }
  
    /// This function can match points in any container that supports
    /// the size() and operator[] methods. The container is usually a
    /// vw::Vector<>, but you could substitute other classes here as
    /// well.
    template <class InterestPointT>
    vw::Matrix<double> operator()(std::vector<InterestPointT> const& ip1,
                                  std::vector<InterestPointT> const& ip2,
                                  vw::Matrix<double> const& /*seed_input*/
                                  = vw::Matrix<double>() ) const {
    
      // check consistency
      VW_ASSERT( ip1.size() == ip2.size(),
                 vw::ArgumentErr() << "Cannot compute fundamental matrix. "
                 << "ip1 and ip2 are not the same size." );
      VW_ASSERT( !ip1.empty() && ip1.size() >= min_elements_needed_for_fit(ip1[0]),
                 vw::ArgumentErr() << "Cannot compute fundamental matrix. "
                 << "Need at at least 8 points, but got: " << ip1.size() << ".\n");

      // Compute the affine fundamental matrix
      Matrix<double> fund = linear_affine_fundamental_matrix(ip1, ip2);

      // Solve for rotation matrices
      double Hl = sqrt(fund(2, 0)*fund(2, 0) + fund(2, 1)*fund(2, 1));
      double Hr = sqrt(fund(0, 2)*fund(0, 2) + fund(1, 2)*fund(1, 2));

      Vector2 epipole(-fund(2, 1), fund(2, 0)), epipole_prime(-fund(1, 2), fund(0, 2));

      if (epipole.x() < 0)
        epipole = -epipole;
      if (epipole_prime.x() < 0)
        epipole_prime = -epipole_prime;
      epipole.y() = -epipole.y();
      epipole_prime.y() = -epipole_prime.y();

      Matrix<double> left_matrix  = math::identity_matrix<3>();
      Matrix<double> right_matrix = math::identity_matrix<3>();
    
      left_matrix(0, 0)  = epipole[0]/Hl;
      left_matrix(0, 1)  = -epipole[1]/Hl;
      left_matrix(1, 0)  = epipole[1]/Hl;
      left_matrix(1, 1)  = epipole[0]/Hl;
      right_matrix(0, 0) = epipole_prime[0]/Hr;
      right_matrix(0, 1) = -epipole_prime[1]/Hr;
      right_matrix(1, 0) = epipole_prime[1]/Hr;
      right_matrix(1, 1) = epipole_prime[0]/Hr;

      // Solve for ideal scaling and translation
      solve_y_scaling(ip1, ip2, left_matrix, right_matrix);

      // Solve for ideal shear, scale, and translation of X axis
      solve_x_shear(ip1, ip2, left_matrix, right_matrix);

      // Work out the ideal render size
      BBox2i left_bbox, right_bbox;
      left_bbox.grow(subvector(left_matrix * Vector3(0,            0,           1), 0, 2));
      left_bbox.grow(subvector(left_matrix * Vector3(m_ldims.x(),  0,           1), 0, 2));
      left_bbox.grow(subvector(left_matrix * Vector3(m_ldims.x(),  m_ldims.y(), 1), 0, 2));
      left_bbox.grow(subvector(left_matrix * Vector3(0,            m_ldims.y(), 1), 0, 2));
      right_bbox.grow(subvector(right_matrix * Vector3(0,            0,           1), 0, 2));
      right_bbox.grow(subvector(right_matrix * Vector3(m_rdims.x(),  0,           1), 0, 2));
      right_bbox.grow(subvector(right_matrix * Vector3(m_rdims.x(),  m_rdims.y(), 1), 0, 2));
      right_bbox.grow(subvector(right_matrix * Vector3(0,            m_rdims.y(), 1), 0, 2));

      // TODO(oalexan1): There is room for improvement below,
      // but the attempts tried below (commented out) need
      // a lot more testing. Also, the current outlier filtering
      // is apparently not foolproof yet.
      
      // Ensure that the transforms map the interest points to points
      // with positive x and y, we will need that when later the
      // transformed images are computed.
      if (m_crop_to_shared_area) 
        left_bbox.crop(right_bbox);
      
      // Note how we subtract left_bbox.min() from both left_matrix
      // and right_matrix.  By subtracting the same thing we
      // maintain the property that a row in the left image is
      // matched to the same row in the right image after the
      // left_matrix and right_matrix transforms are applied.
      left_matrix (0, 2) -= left_bbox.min().x();
      left_matrix (1, 2) -= left_bbox.min().y();
      right_matrix(0, 2) -= left_bbox.min().x();
      right_matrix(1, 2) -= left_bbox.min().y();
      
      // Concatenate these into the answer
      result_type T;
      submatrix(T, 0, 0, 3, 3) = left_matrix;
      submatrix(T, 0, 3, 3, 3) = right_matrix;
      
      // Implicit in the logic below is the fact that left_bbox should now also
      // have left_bbox.min() subtracted from it, after which it becomes the
      // box with lower-left corner being (0, 0) and upper-right corner
      // being (left_bbox.width(), left_bbox.height()) which is
      // what we save here as the upper bound after the transform.
      T(0, 6) = left_bbox.width();
      T(1, 6) = left_bbox.height();

      return T;
    }
  };

  // Find the absolute difference of the y components of the given
  // interest point pair after applying to those points the given
  // epipolar alignment matrices. If these matrices are correct,
  // and the interest point pair is not an outlier, this
  // absolute difference should be close to 0.
  struct EpipolarAlignmentError {
    template <class TransformT, class InterestPointT>
    double operator() (TransformT const& T,
                       InterestPointT const& ip1,
                       InterestPointT const& ip2) const {

      Matrix<double> left_matrix  = submatrix(T, 0, 0, 3, 3);
      Matrix<double> right_matrix = submatrix(T, 0, 3, 3, 3);

      Vector3 L = left_matrix  * Vector3(ip1.x, ip1.y, 1);
      Vector3 R = right_matrix * Vector3(ip2.x, ip2.y, 1);
      double diff = L[1] - R[1];
      return std::abs(diff);
    }
  };

  // Helper function to instantiate a RANSAC class object and immediately call it
  template <class ContainerT1, class ContainerT2, class FittingFuncT, class ErrorFuncT>
  typename FittingFuncT::result_type ransac(std::vector<ContainerT1> const& p1,
                                            std::vector<ContainerT2> const& p2,
                                            FittingFuncT             const& fitting_func,
                                            ErrorFuncT               const& error_func,
                                            int     num_iterations,
                                            double  inlier_threshold,
                                            int     min_num_output_inliers,
                                            bool    reduce_min_num_output_inliers_if_no_fit = false
                                            ) {
    RandomSampleConsensus<FittingFuncT, ErrorFuncT>
      ransac_instance(fitting_func,
                      error_func,
                      num_iterations,
                      inlier_threshold,
                      min_num_output_inliers,
                      reduce_min_num_output_inliers_if_no_fit
                      );
    return ransac_instance(p1,p2);
  }
    
  // Main function that other parts of ASP should use
  Vector2i affine_epipolar_rectification(Vector2i const& left_image_dims,
                                         Vector2i const& right_image_dims,
                                         double inlier_threshold,
                                         int num_ransac_iterations,
                                         std::vector<ip::InterestPoint> const& ip1,
                                         std::vector<ip::InterestPoint> const& ip2,
                                         bool crop_to_shared_area,
                                         Matrix<double>& left_matrix,
                                         Matrix<double>& right_matrix,
                                         // optionally return the inliers
                                         std::vector<size_t> * inliers_ptr) {
  
    int  min_num_output_inliers = ip1.size() / 2;
    bool reduce_min_num_output_inliers_if_no_fit = true;

    vw::Matrix<double> T;
    Stopwatch sw;
    sw.start();

    vw_out() << "Computing the epipolar rectification "
             << "using RANSAC with " << num_ransac_iterations
             << " iterations and inlier threshold " << inlier_threshold << ".\n";

    // If RANSAC fails, it will throw an exception
    BestFitEpipolarAlignment func(left_image_dims, right_image_dims, crop_to_shared_area);
    EpipolarAlignmentError error_metric;
    std::vector<size_t> inlier_indices;
    RandomSampleConsensus<BestFitEpipolarAlignment, EpipolarAlignmentError> 
      ransac(func, error_metric,
             num_ransac_iterations, inlier_threshold,
             min_num_output_inliers, reduce_min_num_output_inliers_if_no_fit);
    
    T = ransac(ip1, ip2);
    inlier_indices = ransac.inlier_indices(T, ip1, ip2);

    vw_out() << "Found " << inlier_indices.size() << " / " << ip1.size() << " inliers.\n";

    sw.stop();
    vw_out(DebugMessage,"asp") << "Elapsed time in computing rectification matrices: "
                               << sw.elapsed_seconds() << " seconds.\n";

    // Extract the matrices and the cropped transformed box from the computed transform
    left_matrix  = submatrix(T, 0, 0, 3, 3);
    right_matrix = submatrix(T, 0, 3, 3, 3);
    Vector2i trans_crop_box(T(0, 6), T(1, 6));

    // Find the maximum error for inliers
    double max_err = 0.0;
    for (size_t it = 0; it < inlier_indices.size(); it++) {
      int i = inlier_indices[it];
      max_err = std::max(max_err, error_metric(T, ip1[i], ip2[i]));
    }
        
    vw_out() << "Maximum absolute difference of y components of "
             << "aligned inlier interest points is "
             << max_err << " pixels." << std::endl;

    // This needs more testing
    if (false && !crop_to_shared_area) {
      // The bounds of the transforms have been a bit too generous. Tighten them to the bounding
      // box of the IP.
      // TODO(oalexan1): Remove outliers here!
      
      // Apply local alignment to inlier ip and estimate the search range
      vw::HomographyTransform left_local_trans (left_matrix);
      vw::HomographyTransform right_local_trans(right_matrix);
      
      // Find the transformed IP
      std::vector<vw::ip::InterestPoint> left_trans_local_ip;
      std::vector<vw::ip::InterestPoint> right_trans_local_ip;

      for (size_t it = 0; it < inlier_indices.size(); it++) {
        int i = inlier_indices[it];
        Vector2 left_pt (ip1[i].x, ip1[i].y);
        Vector2 right_pt(ip2[i].x, ip2[i].y);
        
        left_pt  = left_local_trans.forward(left_pt);
        right_pt = right_local_trans.forward(right_pt);
        
        // First copy all the data from the input ip, then apply the transform
        left_trans_local_ip.push_back(ip1[i]);
        right_trans_local_ip.push_back(ip2[i]);
        left_trans_local_ip.back().x  = left_pt.x();
        left_trans_local_ip.back().y  = left_pt.y();
        right_trans_local_ip.back().x = right_pt.x();
        right_trans_local_ip.back().y = right_pt.y();
      }

      // Filter outliers
      Vector2 params = stereo_settings().outlier_removal_params;
      bool quiet = false;
      if (params[0] < 100.0)
        asp::filter_ip_by_disparity(params[0], params[1], quiet,
                                    left_trans_local_ip, right_trans_local_ip); 
      
      vw::BBox2i left_bbox, right_bbox;
      for (size_t i = 0; i < left_trans_local_ip.size(); i++) {

        Vector2 left_pt (left_trans_local_ip[i].x, left_trans_local_ip[i].y);
        Vector2 right_pt(right_trans_local_ip[i].x, right_trans_local_ip[i].y);
        
        left_bbox.grow(left_pt);
        right_bbox.grow(right_pt);
      }

      // TODO(oalexan1): Run a large scale test to see if this is necessary.
      left_bbox.expand(50);
      right_bbox.expand(50);
      
      // The way the transforms were created, there is no good reason
      // for transformed ip to have negative values.
      left_bbox.min().x() = std::max(left_bbox.min().x(), 0);
      left_bbox.min().y() = std::max(left_bbox.min().y(), 0);
      right_bbox.min().x() = std::max(right_bbox.min().x(), 0);
      right_bbox.min().y() = std::max(right_bbox.min().y(), 0);
      
      // Adjust the domains of the transforms to the bounding boxes of
      // the interest points.
      left_matrix (0, 2) -= left_bbox.min().x();
      left_matrix (1, 2) -= left_bbox.min().y();
      right_matrix(0, 2) -= right_bbox.min().x();
      right_matrix(1, 2) -= right_bbox.min().y();

      trans_crop_box[0] = std::max(left_bbox.width(), right_bbox.width());
      trans_crop_box[1] = std::max(left_bbox.height(), right_bbox.height());
    }
    
    // Optionally return the inliers
    if (inliers_ptr != NULL)
      *inliers_ptr = inlier_indices;
    
    return trans_crop_box;
  }

} // end namespace asp
