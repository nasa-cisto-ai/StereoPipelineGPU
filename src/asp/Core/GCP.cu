#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2006-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <string>
#include <vector>

#include <vw/FileIO/DiskImageView.h>
#include <vw/Image/Interpolation.h>
#include <vw/Cartography/GeoReference.h>
#include <asp/Core/GCP.h>

namespace asp {

// Write a GCP file. Can throw exceptions.
void writeGCP(std::vector<std::string> const& image_files,
              std::string const& gcp_file,
              std::string const& dem_file,
              std::string const& output_prefix,
              asp::MatchList const& matchlist) {
  
  using namespace vw;
  
  // Load a georeference to use for the GCPs from the last image
  vw::cartography::GeoReference georef_image, georef_dem;
  const size_t GEOREF_INDEX = image_files.size() - 1;
  const std::string georef_image_file = image_files[GEOREF_INDEX];
  bool has_georef = vw::cartography::read_georeference(georef_image, georef_image_file);
  // todo(oalexan1): Throw an exception here, then catch it and pop up a message box.
  if (!has_georef)
    vw::vw_throw(vw::ArgumentErr() << "Could not load a valid georeference to use for "
		 << "ground control points in file: " << georef_image_file << ".\n");

  vw::vw_out() << "Loaded georef from file " << georef_image_file << std::endl;
  
  // Init the DEM to use for height interpolation
  boost::shared_ptr<vw::DiskImageResource> dem_rsrc(DiskImageResourcePtr(dem_file));
  vw::DiskImageView<float> dem_disk_image(dem_file);
  vw::ImageViewRef<vw::PixelMask<float>> raw_dem;
  float nodata_val = -std::numeric_limits<float>::max();
  if (dem_rsrc->has_nodata_read()) {
    nodata_val = dem_rsrc->nodata_read();
    raw_dem = vw::create_mask_less_or_equal(dem_disk_image, nodata_val);
  } else {
    raw_dem = vw::pixel_cast<vw::PixelMask<float>>(dem_disk_image);
  }
  vw::PixelMask<float> fill_val;
  fill_val[0] = -99999;
  fill_val.invalidate();
  vw::ImageViewRef<vw::PixelMask<float>> interp_dem
    = vw::interpolate(raw_dem,
                  vw::BilinearInterpolation(),
                  vw::ValueEdgeExtension<vw::PixelMask<float>>(fill_val));
  
  // Load the georef from the DEM
  has_georef = vw::cartography::read_georeference(georef_dem, dem_file);
  if (!has_georef)
    vw::vw_throw(vw::ArgumentErr() << "Could not load a valid georeference from dem file: "
                 << dem_file << ".\n");
  
  vw_out() << "Loaded georef from dem file " << dem_file << std::endl;
  
  BBox2 image_bb = bounding_box(interp_dem);
  vw_out() << "Writing: " << gcp_file << "\n";
  std::ofstream output_handle(gcp_file.c_str());
  output_handle << std::setprecision(17);
  size_t num_pts_skipped = 0, num_pts_used = 0;
  const size_t num_ips    = matchlist.getNumPoints();
  for (size_t p = 0; p < num_ips; p++) { // Loop through IPs
    
    // Compute the GDC coordinate of the point
    ip::InterestPoint ip = matchlist.getPoint(GEOREF_INDEX, p);
    Vector2 lonlat    = georef_image.pixel_to_lonlat(Vector2(ip.x, ip.y));
    Vector2 dem_pixel = georef_dem.lonlat_to_pixel(lonlat);
    PixelMask<float> height = interp_dem(dem_pixel[0], dem_pixel[1])[0];
    
    // We make a separate bounding box check because the ValueEdgeExtension
    //  functionality may not work properly!
    if ( (!image_bb.contains(dem_pixel)) || (!is_valid(height)) ) {
      vw_out() << "Warning: Skipped IP # " << p
               << " because it does not fall on the DEM.\n";
      ++num_pts_skipped;
      continue; // Skip locations which do not fall on the DEM
    }
    
    // Write the per-point information
    output_handle << num_pts_used; // The ground control point ID
    bool write_ecef = false;
    // TODO(oalexan1): It can be convenient to export GCP in ECEF, for software
    // which does not know about projections. Could be an option.
    if (!write_ecef) {
      // Write lat, lon, height
      output_handle << ", " << lonlat[1] << ", " << lonlat[0] << ", " << height[0];
    } else {
      // Write x, y, z
      vw::Vector3 P(lonlat[0], lonlat[1], height[0]);
      P = georef_dem.datum().geodetic_to_cartesian(P);
      output_handle << ", " << P[0] << ' ' << P[1] << ' ' << P[2];
    }
    
    // Write sigma values on the same line
    output_handle << ", " << 1 << ", " << 1 << ", " << 1; 
    
    // Write the per-image information
    // The last image is the reference image, so we skip it when saving GCPs
    size_t num_images = image_files.size();
    size_t num_images_to_save = num_images - 1; 
    for (size_t i = 0; i < num_images_to_save; i++) {
      // Add this IP to the current line
      ip::InterestPoint ip = matchlist.getPoint(i, p);
      output_handle << ", " << image_files[i];
      output_handle << ", " << ip.x << ", " << ip.y; // IP location in image
      output_handle << ", " << 1 << ", " << 1; // Sigma values
    } // End loop through IP sets
    output_handle << std::endl; // Finish the line
    ++num_pts_used;
  } // End loop through IPs
  
  output_handle.close();

  return;
}

} // namespace asp
