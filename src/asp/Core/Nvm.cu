#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Core/Nvm.h>
#include <vw/Core/Exception.h>
#include <vw/Core/Log.h>
#include <vw/FileIO/FileUtils.h>

#include <fstream>
#include <iostream>

namespace asp {

// A wrapper to carry fewer things around
void ReadNVM(std::string const& input_filename, nvmData & nvm) {
  ReadNVM(input_filename,
          &nvm.cid_to_keypoint_map,
          &nvm.cid_to_filename,
          &nvm.pid_to_cid_fid,
          &nvm.pid_to_xyz,
          &nvm.cid_to_cam_t_global);
}
  
// Reads the NVM control network format. The interest points may or may not
// be shifted relative to optical center. The user is responsible for knowing that.
// If a filename having extension _offset.txt instead of .nvm exists, read
// from it the optical center offsets and apply them.
void ReadNVM(std::string const& input_filename,
             std::vector<Eigen::Matrix2Xd> * cid_to_keypoint_map,
             std::vector<std::string> * cid_to_filename,
             std::vector<std::map<int, int>> * pid_to_cid_fid,
             std::vector<Eigen::Vector3d> * pid_to_xyz,
             std::vector<Eigen::Affine3d> * cid_to_cam_t_global) {

  int file_len = input_filename.size(); // cast to int to make subtraction safe
  std::string offset_path = input_filename.substr(0, std::max(file_len - 4, 0)) + "_offsets.txt";
  std::ifstream offset_fh(offset_path.c_str());
  std::map<std::string, Eigen::Vector2d> offsets;
  bool have_offsets = false;
  if (offset_fh.good()) {
    std::cout << "Read and apply optical offsets from: " << offset_path << std::endl;
    std::string name;
    double x, y;
    while (offset_fh >> name >> x >> y) {
      offsets[name] = Eigen::Vector2d(x, y);
    }
    have_offsets = true;
  }
  
  std::ifstream f(input_filename, std::ios::in);
  std::string token;
  std::getline(f, token);
  
  // Assert that we start with our NVM token
  if (token.compare(0, 6, "NVM_V3") != 0) {
    vw::vw_throw(vw::ArgumentErr() << "File doesn't start with NVM token.");
  }

  // Read the number of cameras
  ptrdiff_t number_of_cid;
  f >> number_of_cid;
  if (number_of_cid < 1) {
    vw::vw_throw(vw::ArgumentErr() << "NVM file is missing cameras.");
  }

  // Resize all our structures to support the number of cameras we now expect
  cid_to_keypoint_map->resize(number_of_cid);
  cid_to_filename->resize(number_of_cid);
  cid_to_cam_t_global->resize(number_of_cid);
  for (ptrdiff_t cid = 0; cid < number_of_cid; cid++) {
    // Clear keypoints from map. We'll read these in shortly
    cid_to_keypoint_map->at(cid).resize(Eigen::NoChange_t(), 2);

    // Read the line that contains camera information
    double focal, dist1, dist2;
    Eigen::Quaterniond q;
    Eigen::Vector3d c;
    f >> token >> focal;
    f >> q.w() >> q.x() >> q.y() >> q.z();
    f >> c[0] >> c[1] >> c[2] >> dist1 >> dist2;
    cid_to_filename->at(cid) = token;

    // Solve for t, which is part of the affine transform
    Eigen::Matrix3d r = q.matrix();
    cid_to_cam_t_global->at(cid).linear() = r;
    cid_to_cam_t_global->at(cid).translation() = -r * c;
  }

  // Read the number of points
  ptrdiff_t number_of_pid;
  f >> number_of_pid;
  if (number_of_pid < 1)
    vw::vw_throw(vw::ArgumentErr() << "The NVM file has no triangulated points.");

  // Read the point
  pid_to_cid_fid->resize(number_of_pid);
  pid_to_xyz->resize(number_of_pid);
  Eigen::Vector3d xyz;
  Eigen::Vector3i color;
  Eigen::Vector2d pt;
  ptrdiff_t cid, fid;
  for (ptrdiff_t pid = 0; pid < number_of_pid; pid++) {
    pid_to_cid_fid->at(pid).clear();

    ptrdiff_t number_of_measures;
    f >> xyz[0] >> xyz[1] >> xyz[2] >>
      color[0] >> color[1] >> color[2] >> number_of_measures;
    pid_to_xyz->at(pid) = xyz;
    for (ptrdiff_t m = 0; m < number_of_measures; m++) {
      f >> cid >> fid >> pt[0] >> pt[1];

      if (have_offsets) {
        auto map_it = offsets.find(cid_to_filename->at(cid));
        if (map_it == offsets.end()) {
          vw::vw_throw(vw::ArgumentErr() << "Cannot find optical offset for image "
                       << cid_to_filename->at(cid) << "\n");
        }
        pt[0] += (map_it->second)[0];
        pt[1] += (map_it->second)[1];
      }
      
      pid_to_cid_fid->at(pid)[cid] = fid;

      if (cid_to_keypoint_map->at(cid).cols() <= fid) {
        cid_to_keypoint_map->at(cid).conservativeResize(Eigen::NoChange_t(), fid + 1);
      }
      cid_to_keypoint_map->at(cid).col(fid) = pt;
    }

    if (!f.good())
      vw::vw_throw(vw::ArgumentErr() << "Unable to correctly read PID: " << pid);
  }
}

// Write an nvm file. Note that a single focal length is assumed and no distortion.
// Those are ignored, and only camera poses, matches, and keypoints are used.
void WriteNVM(std::vector<Eigen::Matrix2Xd> const& cid_to_keypoint_map,
              std::vector<std::string> const& cid_to_filename,
              std::vector<double> const& focal_lengths,
              std::vector<std::map<int, int>> const& pid_to_cid_fid,
              std::vector<Eigen::Vector3d> const& pid_to_xyz,
              std::vector<Eigen::Affine3d> const& cid_to_cam_t_global,
              std::string const& output_filename) {

  // Ensure that the output directory having this file exists
  vw::create_out_dir(output_filename);

  vw::vw_out() << "Writing: " << output_filename << std::endl;
  
  std::fstream f(output_filename, std::ios::out);
  f.precision(17); // double precision
  f << "NVM_V3\n";

  if (cid_to_filename.size() != cid_to_keypoint_map.size())
    vw::vw_throw(vw::ArgumentErr() << "Unequal number of filenames and keypoints.");
  if (pid_to_cid_fid.size() != pid_to_xyz.size())
    vw::vw_throw(vw::ArgumentErr() << "Unequal number of pid_to_cid_fid and xyz measurements.");
  if (cid_to_filename.size() != cid_to_cam_t_global.size())
    vw::vw_throw(vw::ArgumentErr() << "Unequal number of filename and camera transforms.");
  
  // Write camera information
  f << cid_to_filename.size() << std::endl;
  for (size_t cid = 0; cid < cid_to_filename.size(); cid++) {

    // World-to-camera rotation quaternion
    Eigen::Quaterniond q(cid_to_cam_t_global[cid].rotation());

    // Camera center in world coordinates
    Eigen::Vector3d t(cid_to_cam_t_global[cid].translation());
    Eigen::Vector3d camera_center =
      - cid_to_cam_t_global[cid].rotation().inverse() * t;

    f << cid_to_filename[cid] << " " << focal_lengths[cid]
      << " " << q.w() << " " << q.x() << " " << q.y() << " " << q.z() << " "
      << camera_center[0] << " " << camera_center[1] << " "
      << camera_center[2] << " " << "0 0\n"; // zero distortion, not used
  }

  // Write the number of points
  f << pid_to_cid_fid.size() << std::endl;

  for (size_t pid = 0; pid < pid_to_cid_fid.size(); pid++) {
    f << pid_to_xyz[pid][0] << " " << pid_to_xyz[pid][1] << " "
      << pid_to_xyz[pid][2] << " 0 0 0 "
      << pid_to_cid_fid[pid].size();

    if (pid_to_cid_fid[pid].size() <= 1)
      vw::vw_throw(vw::ArgumentErr() << "PID " << pid << " has "
                   << pid_to_cid_fid[pid].size() << " measurements.");
    
    for (std::map<int, int>::const_iterator it = pid_to_cid_fid[pid].begin();
         it != pid_to_cid_fid[pid].end(); it++) {
      f << " " << it->first << " " << it->second << " "
        << cid_to_keypoint_map[it->first].col(it->second)[0] << " "
        << cid_to_keypoint_map[it->first].col(it->second)[1];
    }
    f << std::endl;
  }

  // Close the file
  f.flush();
  f.close();
}

} // end namespace asp
