#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file OrthoRasterizer.cc
///
/// Given a point image and corresponding texture, this class
/// resamples the point cloud on a regular grid over the [x,y] plane
/// of the point image; producing an evenly sampled ortho-image with
/// interpolated z values.

#include <vw/Image/ImageView.h>
#include <vw/Image/ImageViewRef.h>
#include <vw/Image/Manipulation.h>
#include <vw/Image/Algorithms.h>
#include <vw/Image/BlockRasterize.h>
#include <vw/Core/ThreadPool.h>
#include <vw/Math/Vector.h>
#include <vw/Math/BBox.h>
#include <vw/Math/Statistics.h>
#include <vw/Image/Filter.h>
#include <vw/Image/InpaintView.h>

#include <asp/Core/SoftwareRenderer.h>
#include <asp/Core/PointUtils.h>
#include <boost/foreach.hpp>
#include <boost/math/special_functions/next.hpp>
#include <asp/Core/OrthoRasterizer.h>
#include <valarray>

namespace asp{

  using namespace vw;

  class compare_bboxes { // simple comparison function
  public:
    bool operator()(const BBox2i A, const BBox2i B) const {
      return ( A.min().x() < B.min().x() );
    }
  };

  void dump_image(std::string const& prefix, BBox2i const& box,
                  ImageViewRef<Vector3> const& I){

    //Crop the image to the given box and save to a file.

    std::ostringstream os;
    os << prefix << "_" << box.min().x() << "_" << box.min().y()
       << " " << box.width() << " " << box.height() << ".csv";
    std::string file = os.str();
    vw_out() << "Writing: " << file << std::endl;
    std::ofstream of(file.c_str());
    of.precision(18);

    BBox2i lbox = box;
    lbox.crop(bounding_box(I));
    ImageView<Vector3> crop_img = crop(I, lbox);
    for (int col = 0; col < crop_img.cols(); col++){
      for (int row = 0; row < crop_img.rows(); row++){
        Vector3 p = crop_img(col, row);
        if (boost::math::isnan(p.z())) continue;
        of << p.x() << ' ' << p.y()  << ' ' << p.z() << std::endl;
      }
    }
    of.close();
  }

  // Task to parallelize the generation of bounding boxes for each block.
  class SubBlockBoundaryTask : public Task, private boost::noncopyable {
    ImageViewRef<Vector3> m_view;
    int    m_sub_block_size;
    BBox2i m_image_bbox;
    BBox3& m_global_bbox;
    std::vector<BBoxPair>& m_point_image_boundaries;
    ImageViewRef<double> const& m_error_image;
    double m_estim_max_error;   // used for outlier removal by tri error based on percentage
    vw::BBox3 m_estim_proj_box; // used for outlier removal in the bounding box computation
    std::vector<double> & m_errors_hist;
    double m_max_valid_triangulation_error; // used for outlier removal based on user thresh
    Mutex& m_mutex;
    const ProgressCallback& m_progress;
    float m_inc_amt;

    // This is growing a bbox of points in point projection and Z
    // values which are altitude.
    struct GrowBBoxAccumulator {
      BBox3 bbox;
      void operator()( Vector3 const& v ) {
        if ( !boost::math::isnan(v.z()) )
          bbox.grow(v);
      }
    };

    struct ErrorHistAccumulator{
      std::vector<double> & m_hist;
      double m_max_val;
      ErrorHistAccumulator(std::vector<double>& hist, double max_val):
        m_hist(hist), m_max_val(max_val){}
      void operator()(double err){
        if (err == 0) return; // null errors come from invalid pixels
        int len = m_hist.size();
        int k = round((len-1)*std::min(err, m_max_val)/m_max_val);
        if (k >= 0 && k < len && m_max_val > 0) {
          // This is a bugfix for an observed situation when err is NaN.
          // In that case the rounding returns a negative integer.
          m_hist[k]++;
        }
      }
    };

  public:
    SubBlockBoundaryTask(ImageViewRef<Vector3> const& view,
                         int sub_block_size,
                         BBox2i const& image_bbox,
                         BBox3       & global_bbox, 
                         std::vector<BBoxPair>& boundaries,
                         ImageViewRef<double> const& error_image,
                         double estim_max_error,
                         vw::BBox3 const& estim_proj_box,
                         std::vector<double> & errors_hist,
                         double max_valid_triangulation_error,
                         Mutex& mutex, const ProgressCallback& progress, float inc_amt ) :
      m_view(view.impl()), m_sub_block_size(sub_block_size),
      m_image_bbox(image_bbox),
      m_global_bbox(global_bbox), m_point_image_boundaries( boundaries ),
      m_error_image(error_image), m_estim_max_error(estim_max_error),
      m_estim_proj_box(estim_proj_box),
      m_errors_hist(errors_hist), m_max_valid_triangulation_error(max_valid_triangulation_error),
      m_mutex( mutex ), m_progress( progress ), m_inc_amt( inc_amt ) {}
      
    void operator()() {
      ImageView<Vector3> local_image = crop(m_view, m_image_bbox);

      bool remove_outliers_with_pct = (!m_errors_hist.empty());
      ImageView<double> local_error;
      if (remove_outliers_with_pct || m_max_valid_triangulation_error > 0.0)
        local_error = crop( m_error_image, m_image_bbox );
      
      // Further subdivide into boundaries so that prerasterize will
      // only query what it needs.
      std::vector<BBox2i> blocks = subdivide_bbox(m_image_bbox, m_sub_block_size, m_sub_block_size);
      BBox3 local_union;
      std::list<BBoxPair> solutions;
      std::vector<double> local_hist(m_errors_hist.size(), 0);
      bool nonempty_estim_proj_box = (!m_estim_proj_box.empty());
      
      for ( size_t i = 0; i < blocks.size(); i++ ) {
        BBox3 pts_bdbox;
        ImageView<Vector3> local_image2 = crop(local_image, blocks[i] - m_image_bbox.min());

        // See if to filter by user-provided m_max_valid_triangulation_error.
        // Otherwise try to make use of m_estim_max_error if it was found.
        // If this is not provided we will later estimate and use such a value automatically
        // when doing the gridding.  
        ImageView<double> local_error2;
        if (m_max_valid_triangulation_error > 0 || m_estim_max_error > 0)
          local_error2 = crop(local_error, blocks[i] - m_image_bbox.min());

        for (int col = 0; col < local_image2.cols(); col++){
          for (int row = 0; row < local_image2.rows(); row++){
            
            // Skip invalid points
            if (boost::math::isnan(local_image2(col, row).z()))
              continue;
            
            // Skip outliers, points not in the estimated bounding box
            if (nonempty_estim_proj_box && !m_estim_proj_box.contains(local_image2(col, row))) {
              continue;
            }
            
            if (m_max_valid_triangulation_error > 0) {
              if (local_error2(col, row) > m_max_valid_triangulation_error)
                continue;
            } else if (m_estim_max_error > 0) {
              if (local_error2(col, row) > m_estim_max_error)
                continue;
            }
            pts_bdbox.grow(local_image2(col, row));
          }
        }
        
        if ( pts_bdbox.min().x() <= pts_bdbox.max().x() &&
             pts_bdbox.min().y() <= pts_bdbox.max().y() ) {
          // pts_bdbox has at least one point. A box of just one
          // point is considered empty by VW. For that reason,
          // grow this box to make it definitely non-empty.
          // Note: for local_union, which will end up contributing
          // to the global bounding box, we don't use the float_next
          // gimmick, as we need the precise box.
          local_union.grow(pts_bdbox);
          // Check for inf, that causes problems
          if (!std::isinf(pts_bdbox.max()[0]) && !std::isinf(pts_bdbox.max()[1])) {
            pts_bdbox.max()[0] = boost::math::float_next(pts_bdbox.max()[0]);
            pts_bdbox.max()[1] = boost::math::float_next(pts_bdbox.max()[1]);
          }
          solutions.push_back( std::make_pair( pts_bdbox, blocks[i] ) );
        }

        if (remove_outliers_with_pct){
          ErrorHistAccumulator error_accum(local_hist, m_estim_max_error);
          for_each_pixel(crop(local_error, blocks[i] - m_image_bbox.min()),
                         error_accum);
        }

      }

      // Append to the global list of boxes and expand the point cloud
      // bounding box.
      if ( local_union != BBox3() ) {
        Mutex::Lock lock( m_mutex );
        for ( std::list<BBoxPair>::const_iterator it = solutions.begin();
              it != solutions.end(); it++ ) {
          m_point_image_boundaries.push_back( *it );
        }

        m_global_bbox.grow( local_union );

        if (remove_outliers_with_pct)
          for (int i = 0; i < (int)m_errors_hist.size(); i++)
            m_errors_hist[i] += local_hist[i];

        m_progress.report_incremental_progress( m_inc_amt );
      }
    }
  }; // End function operator()


  void remove_outliers(ImageView<Vector3> & image, ImageViewRef<double> const& errors,
                       double error_cutoff, BBox2i const& box){

    // Mask as NaN points above triangulation error
    if (error_cutoff < 0)
      return; // nothing to do

    double nan = std::numeric_limits<double>::quiet_NaN();
    ImageView<float> error_copy = crop(errors, box);

    VW_ASSERT(image.cols() == error_copy.cols() &&
              image.rows() == error_copy.rows(),
              ArgumentErr() << "Size mis-match in remove_outliers().");

    for (int col = 0; col < image.cols(); col++){
      for (int row = 0; row < image.rows(); row++){
        if ( error_copy(col, row) > error_cutoff ){
          image(col, row).z() = nan;
        }
      }
    }

  }

  void filter_by_median(ImageView<Vector3> & image, Vector2 const& median_filter_params){

    // If the point cloud height at the current point differs by more
    // than the given threshold from the median of heights in the
    // window of given size centered at the point, remove it as an outlier.

    int    half   = median_filter_params[0]/2; // half window size
    double thresh = median_filter_params[1];
    if (half <= 0 || thresh <= 0)
      return;

    int nc = image.cols(), nr = image.rows(); // shorten
    double nan = std::numeric_limits<double>::quiet_NaN();

    ImageView<Vector3> image_out = copy(image);

    for (int col = 0; col < image.cols(); col++){
      for (int row = 0; row < image.rows(); row++){

        if (boost::math::isnan(image(col, row).z()))
          continue;

        std::vector<double> vals;
        for (int c = std::max(col-half, 0); c <= std::min(col+half, nc-1); c++){
          for (int r = std::max(row-half, 0); r <= std::min(row+half, nr-1); r++){
            if (boost::math::isnan(image(c, r).z()))
              continue;
            vals.push_back(image(c, r).z());
          }
        }
        double median = vw::math::destructive_median(vals);
        if (fabs(median - image(col, row).z()) > thresh){
          image_out(col, row).z() = nan;
        }
      }
    }

    image = copy(image_out);
  }

  // TODO: This function should live somewhere else!
  // Erode this many pixels around invalid pixels
  void erode_image(ImageView<Vector3> & image, int erode_len){

    if (erode_len <= 0) // No erode, we are finished!
      return;

    int    nc      = image.cols(), 
           nr      = image.rows(); // shorten
    double nan     = std::numeric_limits<double>::quiet_NaN();
    int    max_col = nc - 1;
    int    max_row = nr - 1;

    // Create a buffer to modify and a pointer to allow easy
    //  swapping of which image we are writing to.
    // - The first write is to the input image, so we don't have to do extra copies with one pass.
    ImageView<Vector3>  buffer = copy(image);
    ImageView<Vector3>* read_ptr  = &buffer; 
    ImageView<Vector3>* write_ptr = &image; 
    ImageView<Vector3>* temp_ptr = 0;
    
    // One pass per erode length    
    for (int pass = 0; pass < erode_len; pass++){

      // Loop through the entire image
      for (int col = 0; col < nc; col++){
        int start_col = std::max(col-1, 0      );
        int stop_col  = std::min(col+1, max_col);
        for (int row = 0; row < nr; row++){
          int start_row = std::max(row-1, 0      );
          int stop_row  = std::min(row+1, max_row);

          // Loop through bounds checked border 1 region around this pixel
          for (int c = start_col; c <= stop_col; c++){
            for (int r = start_row; r <= stop_row; r++){
              // If any of these pixels are bad, throw out this pixel
              if (boost::math::isnan(read_ptr->operator()(c, r).z()))
                write_ptr->operator()(col, row).z() = nan;
            }
          } // End inner erode double loop
          
        }
      } // End double loop through image pixels

      // Swap the read and write buffer pointers for the next pass
      // - No need to make copies each time, the nan regions will just keep expanding.
      temp_ptr  = write_ptr;
      write_ptr = read_ptr;
      read_ptr  = temp_ptr;
      
    } // end passes
    
    // If we had an even number of erode passes, the last write was to the temp
    //  buffer and we need to copy it back to the input image.
    if ((erode_len % 2) == 0)
      image = copy(buffer);
  }


  // Given a histogram as a vector of counts, based on binning values
  // in the interval [0, max_val] with n bins, find a given percentile
  // error. Here, pct is in [0, 100].
  double percentile_error(std::vector<double> const& hist, double max_val, double pct) {

    double pct_ratio = pct / 100.0;
    
    // Total number of counts in the histogram
    int hist_size = hist.size();
    vw::int64 num_errors = 0;
    for (int s = 0; s < hist_size; s++)
      num_errors += hist[s];

    // The index so that the number of counts up to index is the fraction pct/100.0
    // of the total number of counts.
    
    int cutoff_index = 0;
    vw::int64 sum = 0; // to protect against overflow when adding many numbers
    for (int s = 0; s < hist_size; s++){
      sum += hist[s];
      if (sum >= pct_ratio * num_errors){
        cutoff_index = s;
        break;
      }
    }
    
    return max_val * cutoff_index / double(hist_size);
  }
  
  OrthoRasterizerView::OrthoRasterizerView
  (ImageViewRef<Vector3> point_image, ImageViewRef<double> texture,
   double search_radius_factor, double sigma_factor, bool use_surface_sampling, int pc_tile_size,
   vw::BBox2 const& projwin,
   OutlierRemovalMethod outlier_removal_method,
   Vector2 const& remove_outliers_params,
   ImageViewRef<double> const& error_image,
   double estim_max_error, vw::BBox3 const& estim_proj_box,
   double max_valid_triangulation_error,
   Vector2 median_filter_params, int erode_len, bool has_las_or_csv,
   std::string const& filter,
   double default_grid_size_multiplier,
   std::int64_t * num_invalid_pixels, vw::Mutex *count_mutex,
   const ProgressCallback& progress):
    // Ensure all members are initiated, even if to temporary values
    m_point_image(point_image), m_texture(ImageView<float>(1,1)),
    m_bbox(BBox3()), m_snapped_bbox(BBox3()), m_spacing(0.0), m_default_spacing(0.0),
    m_default_spacing_x(0.0), m_default_spacing_y(0.0),
    m_search_radius_factor(search_radius_factor),
    m_sigma_factor(sigma_factor),
    m_use_surface_sampling(use_surface_sampling),
    m_default_value(0),
    m_minz_as_default(true), m_use_alpha(false),
    m_block_size(pc_tile_size),
    m_projwin(projwin),
    m_error_image(error_image), m_error_cutoff(-1.0),
    m_median_filter_params(median_filter_params), m_erode_len(erode_len),
    m_default_grid_size_multiplier(default_grid_size_multiplier),
    m_num_invalid_pixels(num_invalid_pixels),
    m_count_mutex(count_mutex){

    *m_num_invalid_pixels = 0; // Init counter
    set_texture(texture.impl());

    // Convert the filter from string to enum, to speed up checking against it later
    m_percentile = -1; // ensure it is initialized
    if (filter      == "weighted_average") m_filter = asp::f_weighted_average;
    else if (filter == "min"             ) m_filter = asp::f_min;
    else if (filter == "max"             ) m_filter = asp::f_max;
    else if (filter == "mean"            ) m_filter = asp::f_mean;
    else if (filter == "median"          ) m_filter = asp::f_median;
    else if (filter == "stddev"          ) m_filter = asp::f_stddev;
    else if (filter == "count"           ) m_filter = asp::f_count;
    else if (filter == "nmad"            ) m_filter = asp::f_nmad;
    else if (sscanf (filter.c_str(), "%lf-pct", &m_percentile) == 1)
      m_filter = asp::f_percentile;
    else
    vw_throw( ArgumentErr() << "OrthoRasterize: unknown filter: " << filter << ".\n" );
    
    //dump_image("img", BBox2(0, 0, 3000, 3000), point_image);

    // Compute the bounding box that encompasses tiles within the image
    //
    // They're used for querying what part of the image we need
    VW_OUT(DebugMessage,"asp") << "Computing raster bounding box.\n";

    int num_bins = 1024;
    std::vector<double> errors_hist;
    if (outlier_removal_method != NO_OUTLIER_REMOVAL_METHOD){
      // Need to compute the histogram of all errors in the error image
      errors_hist = std::vector<double>(num_bins, 0.0);
    }

    // Subdivide each block into smaller chunks. Note: small chunks
    // greatly increase the memory usage and run-time for very large
    // images (because they are very many). As such, make the chunks
    // bigger for bigger images.
    double s = 10000.0;
    int sub_block_size
      = int(double(point_image.cols())*double(point_image.rows())/(s*s));
    sub_block_size = std::max(1, sub_block_size);
    sub_block_size = int(round(pow(2.0, floor(log(sub_block_size)/log(2.0)))));
    sub_block_size = std::max(16, sub_block_size);
    sub_block_size = std::min(ASP_MAX_SUBBLOCK_SIZE, sub_block_size);
    std::vector<BBox2i> blocks = subdivide_bbox(m_point_image, m_block_size, m_block_size);

    // Find the bounding box of each subblock, stored in
    // m_point_image_boundaries, together with other info by
    // searching through the image.
    FifoWorkQueue queue( vw_settings().default_num_threads() );
    typedef SubBlockBoundaryTask task_type;
    Mutex mutex;
    float inc_amt = 1.0 / float(blocks.size());
    for ( size_t i = 0; i < blocks.size(); i++ ) {
      boost::shared_ptr<task_type>
        task(new task_type(m_point_image, sub_block_size, blocks[i],
                           m_bbox, m_point_image_boundaries,
                           error_image, estim_max_error, estim_proj_box, errors_hist,
                           max_valid_triangulation_error,
                           mutex, progress, inc_amt));
      queue.add_task(task);
    }
    queue.join_all();
    progress.report_finished();

    if ( m_bbox.empty() )
      vw_throw( ArgumentErr() << "OrthoRasterize: Input point cloud is empty!\n" );

    // Override with user's projwin, if specified
    if (m_projwin != BBox2()){
      subvector(m_bbox.min(), 0, 2) = m_projwin.min();
      subvector(m_bbox.max(), 0, 2) = m_projwin.max();
    }

    VW_OUT(DebugMessage,"asp") << "Point cloud boundary is " << m_bbox << "\n";

    if (outlier_removal_method != NO_OUTLIER_REMOVAL_METHOD) {

      // Per user request, find some error percentiles to print.
      std::map<double, double> percentiles = {{25.0, 0.0}, {50.0, 0.0}, {75.0, 0.0},
                                              // The user-percentile
                                              {remove_outliers_params[0], 0.0}};

      for (auto it = percentiles.begin(); it != percentiles.end(); it++) 
        percentiles[it->first] = percentile_error(errors_hist, estim_max_error,
                                                  it->first);

      vw::int64 num_samples = 0; // to protect against overflow when adding many numbers
      for (int s = 0; s < errors_hist.size(); s++)
        num_samples += errors_hist[s];
      
      // Find the outlier cutoff from the histogram of all errors.
      // The cutoff is the outlier factor times the percentile of the errors.
      double user_percentile = percentiles[remove_outliers_params[0]];

      vw_out() << "Collected a sample of " << num_samples << " positive triangulation errors.\n";
      vw_out() << "Error percentiles: " 
               << "Q1 (25%): " << percentiles[25.0] << ", "
               << "Q2 (50%): " << percentiles[50.0] << ", "
               << "Q3 (75%): " << percentiles[75.0] << "."
               << std::endl;
      
      // Multiply by the outlier factor
      if (outlier_removal_method == PERCENTILE_OUTLIER_METHOD) {
        double factor = remove_outliers_params[1];       // e.g., 3.0
        m_error_cutoff = factor * user_percentile;
        vw_out() << "Computing triangulation error cutoff based on --remove-outliers-params.\n";
      } else if (outlier_removal_method == TUKEY_OUTLIER_METHOD) {
        vw_out() << "Using as outlier cutoff the Tukey formula Q3 + 1.5*(Q3 - Q1)." << std::endl;
        m_error_cutoff = percentiles[75.0] + 1.5*(percentiles[75.0] - percentiles[25.0]);
      } else {
        vw_throw( ArgumentErr() << "Unexpected choice for outlier removal method.\n" );
      }

      vw_out() << "Triangulation error cutoff is " << m_error_cutoff
               << " meters.\n";
      
    }else if (max_valid_triangulation_error > 0.0){
      m_error_cutoff = max_valid_triangulation_error;
      vw_out() << "Manual triangulation error cutoff is " << m_error_cutoff
               << " meters.\n";
    }

    // Find the width and height of the median point cloud pixel in
    // projected coordinates. For las or csv files, this approach
    // does not work.
    int len = m_point_image_boundaries.size();
    if (!has_las_or_csv){
      // This vectors can be large, so don't keep them for too long
      std::vector<double> vx, vy;
      vx.reserve(len); vx.clear();
      vy.reserve(len); vy.clear();
      BOOST_FOREACH( BBoxPair const& boundary, m_point_image_boundaries ) {
        if (boundary.first.empty())
          continue;
        vx.push_back(boundary.first.width() /sub_block_size);
        vy.push_back(boundary.first.height()/sub_block_size);
      }
      std::sort(vx.begin(), vx.end());
      std::sort(vy.begin(), vy.end());
      if (len > 0){
        // Get the median
        // TODO: This is not robust. For lro nac, vertical resolution
        // and horizontal resolution differ by a factor of 4, e.g.,
        // 0.5 m and 2 m. The median can be one of the two, which is
        // wrong.  This code should be an average of the values in the
        // [25%, 75%] range.
        // TODO: Integrate with the logic for mapproject.
        // TODO: The default spacing should be 4x times this.
        // https://github.com/NeoGeographyToolkit/StereoPipeline/issues/173
        m_default_spacing_x = vx[(int)(0.5*len)];
        m_default_spacing_y = vy[(int)(0.5*len)];
      }
    }

    return;
  } // End OrthoRasterizerView Constructor


  // This is kind of like part 2 of the constructor
  // - This function finalizes the spacing and generates a spacing-snapped BBox.
  void OrthoRasterizerView::initialize_spacing(const double spacing) {

    // This must happen after the bounding box was computed, but
    // before setting the spacing.
    if (m_use_surface_sampling){
      // Old way
      BBox3 bbox = m_bbox;
      double bbox_width  = fabs(bbox.max().x() - bbox.min().x());
      double bbox_height = fabs(bbox.max().y() - bbox.min().y());
      double input_image_width  = m_point_image.cols();
      double input_image_height = m_point_image.rows();
      // The formula below is not so good, its output depends strongly
      // on how many rows and columns are in the point cloud.
      m_default_spacing = std::max(bbox_width, bbox_height) / std::max(input_image_width,
                                                                       input_image_height);
    }else{
      // We choose the coarsest of the two spacings
      m_default_spacing = std::max(m_default_spacing_x, m_default_spacing_y);
    }

    // Set the sampling rate (i.e. spacing between pixels)
    this->set_spacing(spacing);
    VW_OUT(DebugMessage,"asp") << "Pixel spacing is " << m_spacing << " pnt/px\n";

    // We will snap the box so that its corners are integer multiples
    // of the grid size. This ensures that any two DEMs
    // with the same grid size and overlapping grids have those
    // grids match perfectly.
    m_snapped_bbox = m_bbox;

    // If the user wants to use m_search_radius_factor to do filling,
    // expand the box to allow the DEM to grow.
    if (m_search_radius_factor > 0) 
      m_snapped_bbox.expand(spacing*m_search_radius_factor);
    
    snap_bbox(m_spacing, m_snapped_bbox);

    // Override with user's projwin, if specified
    if (m_projwin != BBox2()){
      subvector(m_snapped_bbox.min(), 0, 2) = m_projwin.min();
      subvector(m_snapped_bbox.max(), 0, 2) = m_projwin.max();
    }

  } // End function initialize_spacing()

  // Function to convert pixel coordinates to the point domain
  BBox3 OrthoRasterizerView::pixel_to_point_bbox( BBox2 const& inbox ) const {
    BBox3 outbox = m_snapped_bbox;
    int d = (int)m_use_surface_sampling;
    outbox.min().x() = m_snapped_bbox.min().x() + ((double(inbox.min().x() - d))
                                                   * m_spacing);
    outbox.max().x() = m_snapped_bbox.min().x() + ((double(inbox.max().x() - d))
                                                   * m_spacing);
    outbox.min().y() = m_snapped_bbox.min().y() + ((double(rows() - inbox.max().y() - d))
                                                   * m_spacing);
    outbox.max().y() = m_snapped_bbox.min().y() + ((double(rows() - inbox.min().y() - d))
                                                   * m_spacing);
    return outbox;
  }

  /// \cond INTERNAL
  OrthoRasterizerView::prerasterize_type
  OrthoRasterizerView::prerasterize(BBox2i const& bbox) const {

    BBox2i bbox_1 = bbox;
    
    // bugfix, ensure we see enough beyond current tile
    bbox_1.expand((int)ceil(std::max(m_search_radius_factor, 5.0)));

    // Used to find which polygons are actually in the draw space.
    BBox3 local_3d_bbox = pixel_to_point_bbox(bbox_1);

    ImageView<float > render_buffer;
    ImageView<double> d_buffer, weights;
    if (m_use_surface_sampling){
      render_buffer.set_size(bbox_1.width(), bbox_1.height());
    }

    // Setup a software renderer and the orthographic view matrix
    vw::stereo::SoftwareRenderer renderer(bbox_1.width(), bbox_1.height(),
                                          &render_buffer(0,0) );
    renderer.Ortho2D(local_3d_bbox.min().x(), local_3d_bbox.max().x(),
                     local_3d_bbox.min().y(), local_3d_bbox.max().y());

    // Given a DEM grid point, search for cloud points within the
    // circular region of radius equal to grid size. As such, a
    // given cloud point may contribute to multiple DEM points, but
    // with different weights (set by Gaussian). We make this radius
    // no smaller than the default DEM spacing. Search radius can be
    // over-ridden by user.
    double search_radius;
    if (m_search_radius_factor <= 0.0)
      search_radius = std::max(m_spacing, m_default_spacing);
    else
      search_radius = m_spacing*m_search_radius_factor;
    asp::Point2Grid point2grid(bbox_1.width(),
                               bbox_1.height(),
                               d_buffer, weights,
                               local_3d_bbox.min().x(),
                               local_3d_bbox.min().y(),
                               m_spacing, m_default_spacing,
                               search_radius, m_sigma_factor,
                               m_filter, m_percentile);
    
    // Set up the default color value
    double min_val = 0.0;
    if (m_use_alpha) {
      // use this dummy value to denote transparency
      min_val = std::numeric_limits<float>::min();
    } else if (m_minz_as_default) {
      min_val = m_snapped_bbox.min().z();
    } else {
      min_val = m_default_value;
    }

    std::valarray<float> vertices(10), intensities(5);

    if (m_use_surface_sampling){
      static const int NUM_COLOR_COMPONENTS  = 1;  // We only need gray scale
      static const int NUM_VERTEX_COMPONENTS = 2; // DEMs are 2D
      renderer.Clear(min_val);
      renderer.SetVertexPointer(NUM_VERTEX_COMPONENTS, &vertices[0]);
      renderer.SetColorPointer(NUM_COLOR_COMPONENTS, &intensities[0]);
    }else{
      point2grid.Clear(min_val);
    }

    // For each block in the DEM space intersecting local_3d_bbox,
    // find the corresponding blocks in the point cloud space.  We
    // use here a map since we'd like to group together the point
    // cloud blocks which fall within the same 256 x 256 tile, to do
    // their union instead of them individually, for reasons of speed.
    typedef std::map<BBox2i, BBox2i, compare_bboxes> BlockMapType;
    typedef BlockMapType::iterator MapIterType;
    BlockMapType blocks_map;
    BOOST_FOREACH( BBoxPair const& boundary, m_point_image_boundaries ) {
      if (! local_3d_bbox.intersects(boundary.first) )
        continue;

      BBox2i pc_block = boundary.second;

      BBox2i snapped_block;
      snapped_block.min() = m_block_size*floor(pc_block.min()/double(m_block_size));
      snapped_block.max() = m_block_size*ceil( pc_block.max()/double(m_block_size));
      MapIterType it = blocks_map.find(snapped_block);
      if (it != blocks_map.end() ){
        (it->second).grow(pc_block);
      }else{
        blocks_map.insert(std::pair<BBox2, BBox2>(snapped_block, pc_block));
      }

    }

    if ( blocks_map.empty() ){
      // TODO: Don't include these pixels in the total?
      { // Lock and update the total number of invalid pixels in this tile.
        vw::Mutex::Lock lock(*m_count_mutex);
        // Care here, convert to int64_t before multiplication, to avoid
        // int32 overflow.
        (*m_num_invalid_pixels) += std::int64_t(bbox.width())*std::int64_t(bbox.height());
      }
      
      if (m_use_surface_sampling){
        return prerasterize_type(render_buffer, BBox2i(-bbox_1.min().x(),
                                                       -bbox_1.min().y(), cols(), rows()));
      }else{
        return prerasterize_type(d_buffer, BBox2i(-bbox_1.min().x(),
                                                  -bbox_1.min().y(), cols(), rows()) );
      }
    }

    // This is very important. When doing surface sampling, for each
    // pixel we need to see its next up and right neighbors.
    int d = (int)m_use_surface_sampling;

    for (MapIterType it = blocks_map.begin(); it != blocks_map.end(); it++){

      BBox2i block = it->second;

      block.max() += Vector2i(d, d);
      block.crop(vw::bounding_box(m_point_image));

      // Pull a copy of the input image in memory.  Expand the image
      // to be able to see a bit beyond when filling holes.
      BBox2i biased_block = block;
      int bias = m_median_filter_params[0]/2 + m_erode_len;
      biased_block.expand(bias);
      biased_block.crop(vw::bounding_box(m_point_image));
      ImageView<Vector3> point_copy = crop(m_point_image, biased_block);

      remove_outliers(point_copy, m_error_image, m_error_cutoff, biased_block);
      filter_by_median(point_copy, m_median_filter_params);
      erode_image(point_copy, m_erode_len);

      // Crop back to the area of interest
      point_copy = crop(point_copy, block - biased_block.min());

      ImageView<float> texture_copy = crop(m_texture, block );

      typedef ImageView<Vector3>::pixel_accessor PointAcc;
      PointAcc row_acc = point_copy.origin();
      for ( int32 row = 0; row < point_copy.rows()-d; ++row ) {
        PointAcc point_ul = row_acc;

        for ( int32 col = 0; col < point_copy.cols()-d; ++col ) {

          PointAcc point_ur = point_ul; point_ur.next_col();
          PointAcc point_ll = point_ul; point_ll.next_row();
          PointAcc point_lr = point_ul; point_lr.advance(1,1);

          if (m_use_surface_sampling){

            // This loop rasterizes a quad indexed by the upper left.
            if ( !boost::math::isnan((*point_ul).z()) &&
                 !boost::math::isnan((*point_lr).z()) ) {

              vertices[0] = (*point_ul).x(); // UL
              vertices[1] = (*point_ul).y();
              vertices[2] = (*point_ll).x(); // LL
              vertices[3] = (*point_ll).y();
              vertices[4] = (*point_lr).x(); // LR
              vertices[5] = (*point_lr).y();
              vertices[6] = (*point_ur).x(); // UR
              vertices[7] = (*point_ur).y();
              vertices[8] = (*point_ul).x(); // UL
              vertices[9] = (*point_ul).y();

              intensities[0] = texture_copy(col,  row);
              intensities[1] = texture_copy(col,row+1);
              intensities[2] = texture_copy(col+1,  row+1);
              intensities[3] = texture_copy(col+1,row);
              intensities[4] = texture_copy(col,row);

              if (!boost::math::isnan((*point_ll).z())) {
                // triangle 1 is: UL LL LR
                renderer.DrawPolygon(0, 3);
              }
              if (!boost::math::isnan((*point_ur).z())) {
                // triangle 2 is: LR, UR, UL
                renderer.DrawPolygon(2, 3);
              }
            }

          }else{
            // The new engine
            if ( !boost::math::isnan(point_copy(col, row).z()) &&
                 local_3d_bbox.contains(point_copy(col, row))){
              point2grid.AddPoint(point_copy(col, row).x(),
                                  point_copy(col, row).y(),
                                  texture_copy(col,  row));
            }
          }
          point_ul.next_col();
        } // End column loop
        row_acc.next_row();
      } // End row loop

    }

    if (!m_use_surface_sampling)
      point2grid.normalize();

    // The software renderer returns an image which will render
    // upside down in most image formats, so we correct that here.
    // We also introduce transparent pixels into the result where necessary.
    // TODO: Here can do flipping in place.
    ImageView<PixelGray<float>> result;
    if (m_use_surface_sampling)
      result = flip_vertical(render_buffer);
    else
      result = flip_vertical(d_buffer);

    // Loop through result here and count up how many pixels have been
    // changed from the default value.
    std::int64_t num_unset = 0;
    for (int r=0; r<result.rows(); ++r) {
      for (int c=0; c<result.cols(); ++c) {
        
        Vector2i pix = Vector2(c, r) + bbox_1.min();
        if (bbox.contains(pix)) {
          //  Ignore the pixels in the temporary extension of bbox.
          if (result(c,r) == min_val)
            ++num_unset;
        }
      }
    }
    { // Lock and update the total number of invalid pixels in this tile.
      vw::Mutex::Lock lock(*m_count_mutex);
      (*m_num_invalid_pixels) += num_unset;
    }

    return prerasterize_type(result,
                             BBox2i(-bbox_1.min().x(), -bbox_1.min().y(), cols(), rows()));
  }

  // Return the affine georeferencing transform.
  vw::Matrix<double,3,3> OrthoRasterizerView::geo_transform() {
    vw::Matrix<double,3,3> geo_transform;
    geo_transform.set_identity();
    geo_transform(0,0) = m_spacing;
    geo_transform(1,1) = -m_spacing;
    geo_transform(0,2) = m_snapped_bbox.min().x();
    geo_transform(1,2) = m_snapped_bbox.max().y();
    return geo_transform;
  }

} // namespace asp
