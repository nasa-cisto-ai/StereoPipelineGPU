#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Core/Bathymetry.h>
#include <vw/Camera/CameraModel.h>
#include <vw/Core/Exception.h>
#include <vw/Math/LevenbergMarquardt.h>

// Turn this on to verify if Snell's law holds given the incoming and
// outgoing rays to a plane. Then stereo_tri needs to be run with one
// thread as this will print a lot of text.
#define DEBUG_BATHY 0

namespace asp {

  using namespace vw;
  using namespace vw::stereo;

  void read_bathy_plane(std::string const& bathy_plane_file,
                        std::vector<double> & bathy_plane,
                        bool & use_curved_water_surface,
                        vw::cartography::GeoReference & water_surface_projection) {

    vw_out() << "Reading the water surface from: " << bathy_plane_file << "\n";
    
    std::ifstream handle;
    handle.open(bathy_plane_file.c_str());
    if (handle.fail()) 
      vw_throw(vw::IOErr() << "Unable to open bathy plane file: " << bathy_plane_file << "\n");
    
    std::vector<std::string> lines;
    std::string line;
    while ( getline(handle, line, '\n') ){
      lines.push_back(line);
    }

    if (lines.empty())
      vw_throw(vw::IOErr() << "Invalid bathy plane file: " << bathy_plane_file << "\n");
      
    // Read the four values
    bathy_plane.resize(4);
    {
      std::istringstream iss(lines[0]);
      if (!(iss >> bathy_plane[0] >> bathy_plane[1] >> bathy_plane[2] >> bathy_plane[3])) 
        vw_throw(vw::IOErr() << "Could not read four values from first "
                 << "line of the bathy plane.\n");
    }
    
    // The second line must start with a comment for the plane to be a curved surface
    if (lines.size() <= 1 || lines[1][0] != '#')
      use_curved_water_surface = false;
    else 
     use_curved_water_surface = true;
    
    if (!use_curved_water_surface && bathy_plane[3] >= 0) 
      vw_throw(vw::IOErr() << "For a planar water surface, the fourth value "
               << "must be negative.\n");

    if (!use_curved_water_surface)
      return; // nothing more to do in this case
    
    if (bathy_plane[2] <= 0) 
      vw_throw(vw::IOErr() << "For a curved water surface, the third value "
               << "must be positive.\n");

    double scale = 1.0;
    double proj_lat, proj_lon;
    {
      if (lines.size() == 2)
        vw_throw(vw::IOErr() << "Invalid bathy plane file: " << bathy_plane_file << "\n");

      std::istringstream iss(lines[2]);
      if (!(iss >> proj_lat >> proj_lon)) 
        vw_throw(vw::IOErr() << "Could not read the projection latitude and longitude.\n");
    }
    vw::cartography::Datum datum("WGS_1984");
    water_surface_projection.set_datum(datum);
    water_surface_projection.set_stereographic(proj_lat, proj_lon, scale);
    vw_out() << "Read projection: " <<  water_surface_projection.overall_proj4_str()
             << std::endl;
  }

  // Read left and right bathy plane settings and associated data.
  // More often than not they will be identical.
  void read_bathy_plane_set(std::string const& bathy_plane_files,
                            std::vector<BathyPlaneSettings> & bathy_plane_set) {

    bathy_plane_set.clear();
    
    std::string bathy_plane_file;
    std::istringstream iss(bathy_plane_files);
    while (iss >> bathy_plane_file) {

      bathy_plane_set.push_back(BathyPlaneSettings());
      read_bathy_plane(bathy_plane_file,
                       bathy_plane_set.back().bathy_plane,  
                       bathy_plane_set.back().use_curved_water_surface,  
                       bathy_plane_set.back().water_surface_projection);
    }

    if (bathy_plane_set.size() != 1 && bathy_plane_set.size() != 2) 
      vw_throw(vw::ArgumentErr() << "One or two bathy planes expected.\n");

    // Clone the bathy plane if there's only one
    if (bathy_plane_set.size() == 1)
      bathy_plane_set.push_back(bathy_plane_set[0]);
  }
    
  // Given a plane as four values a, b, c, d, with the plane being a * x + b * y + c * z + d = 0,
  // find how far off a point (x, y, z) is from the plane by evaluating the above expression.
  double signed_dist_to_plane(std::vector<double> const& plane, vw::Vector3 const& point) {

    double ans = 0.0;
    for (unsigned coord_it = 0; coord_it < 3; coord_it++) {
      ans += plane[coord_it] * point[coord_it];
    }
    ans += plane[3];
    
    return ans;
  }

  // Compute the projected coordinates of an ECEF point
  inline Vector3 proj_point(vw::cartography::GeoReference const& projection,
                            Vector3 const& xyz) {
    return projection.geodetic_to_point(projection.datum().cartesian_to_geodetic(xyz));
  }

  // Reverse this operation
  inline Vector3 unproj_point(vw::cartography::GeoReference const& projection,
                              Vector3 const& proj_pt) {
    return projection.datum().geodetic_to_cartesian(projection.point_to_geodetic(proj_pt));
  }

  // Given a ECEF point xyz, and two planes, find if xyz is above or below each of the
  // plane by finding the signed distances to them.
  void signed_distances_to_planes(bool use_curved_water_surface,
                                  std::vector<BathyPlaneSettings> const& bathy_set,
                                  vw::Vector3 const& xyz,
                                  std::vector<double> & distances) {
    
    if (bathy_set.size() != 2) 
      vw_throw(vw::ArgumentErr() << "Two bathy planes expected.\n");
    
    distances.resize(2);
    for (size_t it = 0; it < 2; it++) {
      // For a curved water surface need to first convert xyz to projected coordinates
      if (use_curved_water_surface)
        distances[it] = signed_dist_to_plane(bathy_set[it].bathy_plane,
                                             proj_point(bathy_set[it].water_surface_projection,
                                                        xyz));
      else
        distances[it] = signed_dist_to_plane(bathy_set[it].bathy_plane, xyz);
    }
  }

  // Test Snell's law in projected and unprojected coordinates
  void test_snells_law(std::vector<double> const& plane,
                       vw::cartography::GeoReference const& water_surface_projection,
                       double refraction_index,
                       vw::Vector3 const& out_unproj_xyz,
                       vw::Vector3 const& in_unproj_dir, vw::Vector3 const& out_unproj_dir,
                       vw::Vector3 const& out_proj_xyz, 
                       vw::Vector3 const& in_proj_dir, vw::Vector3 const& out_proj_dir) {
  
    // Verify Snell's law.
    
    // 1. In projected coordinates
    Vector3 proj_normal(plane[0], plane[1], plane[2]);
    double sin_in = sin(acos(dot_prod(proj_normal, -in_proj_dir)));
    double sin_out = sin(acos(dot_prod(-proj_normal, out_proj_dir)));
    std::cout << "proj sin_in, sin_out, sin_in - index * sin_out "
              << sin_in << ' ' << sin_out << ' '
              << sin_in - refraction_index * sin_out << std::endl;
  
    // 2. In unprojected coordinates
    Vector3 proj_xyz_above_normal = out_proj_xyz + 1.0 * proj_normal; // go 1 m along the normal
    Vector3 unproj_xyz_above_normal = unproj_point(water_surface_projection,
                                                   proj_xyz_above_normal);
    Vector3 unproj_normal = unproj_xyz_above_normal - out_unproj_xyz;
    unproj_normal /= norm_2(unproj_normal); // normalize
    sin_in = sin(acos(dot_prod(unproj_normal, -in_unproj_dir)));
    sin_out = sin(acos(dot_prod(-unproj_normal, out_unproj_dir)));
    std::cout << "unproj sin_in, sin_out, sin_in - index * sin_out "
              << sin_in << ' ' << sin_out << ' '
              << sin_in - refraction_index * sin_out << std::endl;
  
    // Verify that the incoming ray, outgoing ray, and the
    // normal are in the same plane in projected coordinates
  
    // 1. In projected coordinates
    Vector3 in_out_normal = vw::math::cross_prod(in_proj_dir, out_proj_dir);
    double plane_error = dot_prod(in_out_normal, proj_normal);
    std::cout << "proj plane error " << plane_error << std::endl;
  
    // 2. In unprojected coordinates
    in_out_normal = vw::math::cross_prod(in_unproj_dir, out_unproj_dir);
    plane_error = dot_prod(in_out_normal, unproj_normal);
    std::cout << "unproj plane error " << plane_error << std::endl;
  
    std::cout << std::endl;
  }

  // See the .h file for more info
  bool snells_law(Vector3 const& in_xyz, Vector3 const& in_dir,
                  std::vector<double> const& plane,
                  double refraction_index, 
                  Vector3 & out_xyz, Vector3 & out_dir) {

    // The ray is given as in_xyz + alpha * in_dir, where alpha is real.
    // See where it intersects the plane.
    double cn = 0.0, dn = 0.0; // Dot product of in_xyz and in_dir with plane normal n
    for (size_t it = 0; it < 3; it++) {
      cn += plane[it] * in_xyz[it];
      dn += plane[it] * in_dir[it];
    }

    // The ray must descend to the plane, or else something is not right
    if (dn >= 0.0)
      return false;
    
    double alpha = -(plane[3] + cn)/dn;
  
    // The intersection with the plane
    out_xyz = in_xyz + alpha * in_dir;

    // Let n be the plane normal pointing up (the first three components
    // of the plane vector). Let out_dir be the outgoing vector after the
    // ray hits the water, according to Snell's law, with in_dir being the
    // incoming ray. Let a1 be the angles between -in_dir and n, a2 be the
    // angle between out_dir and -n.
  
    // Then sin(a1) = refraction_index * sin(a2) per Snell's law.
    // Square this. Note that cos^2 (x) + sin^2 (x) = 1.
    // So, 1 - cos(a1)^2 = refraction_index^2 * (1 - cos(a2)^2).
    // But cos(a1) = dot_product(-in_dir, n) = -dn.
    // So, cos(a2)^2 = 1 - (1 - dn^2)/refraction_index^2
    // Call the left-hand value cos_sq.

    double cos_sq = 1.0 - (1.0 - dn * dn)/refraction_index/refraction_index;
  
    // The outgoing vector out_dir will be a linear combination of -n and d1,
    // normalized to unit length. Let alpha > 0 be the value which will
    // produce the linear combination.  So,
    // out_dir = (-n + alpha * in_dir)/norm(-n + alpha * in_dir)
    // But dot(out_dir, -n) = cos(a2). Hence, if we dot the above with n and square it,
    // we get 
    // cos(a2)^2 = (-1 + alpha * dn)^2 / dot( -n + alpha * in_dir, -n + alpha * in_dir)
    // or 
    // cos(a2)^2 * dot( -n + alpha * in_dir, -n + alpha * in_dir) = (-1 + alpha * dn)^2  
    // or
    // cos_sq * (1 - 2 * alpha * dn + alpha^2) = ( 1 - 2*alpha * dn + alpha^2 * dn^2)
    //
    // Note that we computed cos_sq from Snell's law above.
  
    // Move everything to the left and find the coefficients of the
    // quadratic equation in alpha, so u * alpha^2 + v * alpha + w = 0.
    double u = cos_sq - dn * dn;  // this is cos(a2)^2 - cos(a1)^2 > 0 as a2 < a1
    double v = -2 * dn * cos_sq + 2.0 * dn;
    double w = cos_sq - 1.0;
    double delta = v * v - 4 * u * w; // discriminant
    if (u <= 0.0 || delta < 0.0) 
      return false; // must not happen
  
    alpha = (-v + sqrt(delta)) / (2.0 * u); // pick the positive quadratic root

    if (alpha < 0) 
      return false; // must not happen
    
    // The normalized direction after the ray is bent
    out_dir = -Vector3(plane[0], plane[1], plane[2]) + alpha * in_dir;
    out_dir = out_dir / norm_2(out_dir);

    return true;
  }  

  // Consider a stereographic projection and a plane
  // a * x + b * y + c * z + d = 0 for (x, y, z) in this projection.
  // Intersect it with a ray given in ECEF coordinates.
  // If the values a and b are 0, that is the same as intersecting
  // the ray with the spheroid of values -d/c above the datum.
  // This solver was not used as it was too slow. An approximate
  // solution was instead found.
  class SolveCurvedPlaneIntersection:
    public vw::math::LeastSquaresModelBase<SolveCurvedPlaneIntersection> {
    vw::Vector3 const& m_ray_pt;
    vw::Vector3 const& m_ray_dir;
    vw::cartography::GeoReference const& m_projection;
    std::vector<double> const& m_proj_plane;
  public:

    // This is a one-parameter problem, yet have to use a vector (of size 1)
    // as required by the API.
    typedef vw::Vector<double, 1> result_type;   // residual
    typedef vw::Vector<double, 1> domain_type;   // parameter giving the position on the ray
    typedef vw::Matrix<double>    jacobian_type;

    /// Instantiate the solver with a set of xyz to pixel pairs and a pinhole model
    SolveCurvedPlaneIntersection(vw::Vector3 const& ray_pt, vw::Vector3 const& ray_dir,
                                 vw::cartography::GeoReference const& projection,
                                 std::vector<double> const& proj_plane):
      m_ray_pt(ray_pt), m_ray_dir(ray_dir), m_projection(projection), m_proj_plane(proj_plane) {}

    /// Given the camera, project xyz into it
    inline result_type operator()(domain_type const& t) const {

      // Get the current point along the ray
      Vector3 xyz = m_ray_pt + t[0] * m_ray_dir;

      // Convert to projected coordinates
      Vector3 proj_pt = proj_point(m_projection, xyz);

      result_type ans;
      ans[0] = signed_dist_to_plane(m_proj_plane, proj_pt);
      return ans;
    }
  }; // End class SolveCurvedPlaneIntersection

  // Given a ray in ECEF and a water surface which is a plane only in
  // a local stereographic projection, compute how the ray bends under
  // Snell's law. Use the following approximate logic. Find where the
  // ray intersects the datum with the mean water height, as then it
  // is close to the water surface, since the water surface is almost
  // horizontal in projected coordinates. Find a point on that ray 1 m
  // before that. Convert both of these points from ECEF to the
  // projected coordinate system. Do Snell's law in that coordinate
  // system for the ray going through those two projected points. Find
  // a point on the outgoing ray in projected coordinates Find another
  // close point further along it. Undo the projection for these two
  // points. That will give the outgoing direction in ECEF.
  bool snells_law_curved(Vector3 const& in_xyz, Vector3 const& in_dir,
                         std::vector<double> const& plane,
                         vw::cartography::GeoReference const& water_surface_projection,
                         double refraction_index, 
                         Vector3 & out_xyz, Vector3 & out_dir) {
        
    // Find the mean water surface
    double mean_ht = -plane[3] / plane[2];
    double major_radius
      = water_surface_projection.datum().semi_major_axis() + mean_ht;
    double minor_radius
      = water_surface_projection.datum().semi_minor_axis() + mean_ht;
          
    // Intersect the ray with the mean water surface, this will
    // give us the initial guess for intersecting with that
    // surface. The precise value of this is not important, as
    // long as it is rather close to the plane and on that ray.
    Vector3 guess_xyz = vw::cartography::datum_intersection(major_radius, minor_radius,
                                                            in_xyz, in_dir);

    // Move a little up the ray
    Vector3 prev_xyz = guess_xyz - 1.0 * in_dir;
          
    Vector3 in_proj_xyz = proj_point(water_surface_projection, guess_xyz);
    Vector3 prev_proj_xyz = proj_point(water_surface_projection, prev_xyz);
          
    Vector3 in_proj_dir = in_proj_xyz - prev_proj_xyz;
    in_proj_dir /= norm_2(in_proj_dir);

    // Snell's law in projected coordinates
    Vector3 out_proj_xyz, out_proj_dir;
    bool ans = snells_law(in_proj_xyz, in_proj_dir,
                          plane, refraction_index,
                          out_proj_xyz, out_proj_dir);

    // If Snell's law failed to work, exit early
    if (!ans)
      return ans;
          
    Vector3 next_proj_xyz = out_proj_xyz + 1.0 * out_proj_dir;

    // Convert back to ECEF
    out_xyz = unproj_point(water_surface_projection, out_proj_xyz);
    Vector3 next_xyz = unproj_point(water_surface_projection, next_proj_xyz);

    // Finally get the outgoing direction according to Snell's law in ECEF
    out_dir = next_xyz - out_xyz;
    out_dir /= norm_2(out_dir);

#if 0
    // Refine out_xyz with a solver, with 1e-16 tolerance, after using
    // that 1 m perturbation, as above. The parameter determining the
    // position on the ray changes by under 2.6e-8, so it is not worth
    // it. It is rather slow too.  Then one would need to still
    // recompute out_dir somehow, if doing things this way.
    SolveCurvedPlaneIntersection model(in_xyz, in_dir, water_surface_projection, plane);
    vw::Vector<double> objective(1), start(1);
    start[0] = norm_2(out_xyz - in_xyz);
    objective[0] = 0.0; 
    int status = -1; // will change
    Vector<double> solution = math::levenberg_marquardt(model, start, objective, status);

    out_xyz = in_xyz + solution[0] * in_dir; 
#endif
    
#if 0
    // Sanity check
    test_snells_law(plane,  
                    water_surface_projection,  
                    refraction_index,
                    out_xyz, in_dir, out_dir,
                    out_proj_xyz, in_proj_dir, out_proj_dir);
#endif

    return true;
  }
  
  // Settings used for bathymetry correction
  void BathyStereoModel::set_bathy(double refraction_index,
                                   std::vector<BathyPlaneSettings> const& bathy_set) {
    
    m_bathy_correct = true;
    m_refraction_index = refraction_index;
    m_bathy_set = bathy_set;
    
    if (m_refraction_index <= 1) 
      vw::vw_throw(vw::ArgumentErr() << "The water refraction index must be bigger than 1.");

    if (m_bathy_set.size() != 2) 
      vw::vw_throw(vw::ArgumentErr() << "Expecting two bathy planes (left and right).");

    for (int it = 0; it < 2; it++) {
      if (m_bathy_set[it].bathy_plane.size() != 4)
        vw::vw_throw(vw::ArgumentErr() << "The bathy plane must have 4 coefficients.");
    }

    if (m_bathy_set[0].use_curved_water_surface != m_bathy_set[1].use_curved_water_surface)
      vw::vw_throw(vw::ArgumentErr()
                   << "Either both or none of the bathy planes must model the "
                   << "curvature of the water surface.");

    // The default behavior is for the left and right bathy planes to be the same.
    // Yet we allow them to be different. Here need to check.
    m_single_bathy_plane = true;
    if (m_bathy_set[0].use_curved_water_surface != m_bathy_set[1].use_curved_water_surface)
      m_single_bathy_plane = false;
    if (m_bathy_set[0].water_surface_projection.proj4_str()
        != m_bathy_set[1].water_surface_projection.proj4_str())
       m_single_bathy_plane = false;
    if (m_bathy_set[0].bathy_plane != m_bathy_set[1].bathy_plane)
      m_single_bathy_plane = false;
  }

  // Compute the rays intersection. Note that even if we are in
  // bathymetry mode, so m_bathy_correct is true, for this particular
  // pair of rays we may have do_bathy false, and then we won't do the
  // correction.  Return also a flag saying if we did bathymetry
  // correction or not.
  Vector3 BathyStereoModel::operator()(std::vector<Vector2> const& pixVec,
                                       Vector3& errorVec, bool do_bathy,
                                       bool & did_bathy) const {
  
    // Initialize the outputs
    did_bathy = false;
    errorVec = Vector3();
    
    // It was verified beforehand that both bathy planes have the same
    // value for use_curved_water_surface.
    bool use_curved_water_surface = m_bathy_set[0].use_curved_water_surface;
    
    int num_cams = m_cameras.size();
    VW_ASSERT((int)pixVec.size() == num_cams,
              vw::ArgumentErr() << "the number of rays must match "
              << "the number of cameras.\n");
  
    try {

      std::vector<Vector3> camDirs(num_cams), camCtrs(num_cams);
      camDirs.clear(); camCtrs.clear();
    
      // Pick the valid rays
      for (int p = 0; p < num_cams; p++){
      
        Vector2 pix = pixVec[p];
        if (pix != pix || // i.e., NaN
            pix == camera::CameraModel::invalid_pixel() ) 
          continue;
      
        camDirs.push_back(m_cameras[p]->pixel_to_vector(pix));
        camCtrs.push_back(m_cameras[p]->camera_center(pix));
      }

      // Not enough valid rays
      if (camDirs.size() < 2) 
        return Vector3();

      if (are_nearly_parallel(m_least_squares, m_angle_tol, camDirs)) 
        return Vector3();

      // Determine range by triangulation
      Vector3 uncorr_tri_pt = triangulate_point(camDirs, camCtrs, errorVec);
      if ( m_least_squares ){
        if (num_cams == 2)
          refine_point(pixVec[0], pixVec[1], uncorr_tri_pt);
        else
          vw::vw_throw(vw::NoImplErr() << "Least squares refinement is not "
                       << "implemented for multi-view stereo.");
      }
    
      // Reflect points that fall behind one of the two cameras.  Do
      // not do this when bathymetry mode is on, as then we surely
      // have satellite images and there is no way a point would be
      // behind the camera.
      if (!m_bathy_correct) {
        bool reflect = false;
        for (int p = 0; p < (int)camCtrs.size(); p++)
          if (dot_prod(uncorr_tri_pt - camCtrs[p], camDirs[p]) < 0)
            reflect = true;
        if (reflect)
          uncorr_tri_pt = -uncorr_tri_pt + 2*camCtrs[0];
      }

      if (!do_bathy || camDirs.size() != 2) 
        return uncorr_tri_pt;
    
      // Continue with bathymetry correction
      
      if (!m_bathy_correct) 
        vw::vw_throw(vw::ArgumentErr()
                     << "Requested to do bathymetry correction while "
                     << "this mode was not set up.");

      // Find the rays after bending, according to Snell's law.
      std::vector<Vector3> waterDirs(2), waterCtrs(2);

      // When there's a single plane, things are simple.
      // Rays get bent or not, then they intersect, and done.
      if (m_single_bathy_plane) {
      
        if (!use_curved_water_surface) {
          
          double ht_val = signed_dist_to_plane(m_bathy_set[0].bathy_plane, uncorr_tri_pt);
          if (ht_val >= 0) {
            // the rays intersect above the water surface, no need to go on
            did_bathy = false;
            return uncorr_tri_pt;
          }
          
          // The simple case, when the water surface is a plane in ECEF
          for (size_t it = 0; it < 2; it++) {
            bool ans = snells_law(camCtrs[it], camDirs[it], m_bathy_set[it].bathy_plane,
                                  m_refraction_index, 
                                  waterCtrs[it], waterDirs[it]);
            // If Snell's law failed to work, return the result before it
            if (!ans) {
              did_bathy = false;
              return uncorr_tri_pt;
            }
          }
          
        } else{
          
          // The more complex case, the water surface is curved. It is
          // however flat (a plane) if we switch to proj coordinates.
          Vector3 proj_pt = proj_point(m_bathy_set[0].water_surface_projection, uncorr_tri_pt);
          double ht_val = signed_dist_to_plane(m_bathy_set[0].bathy_plane, proj_pt);
          if (ht_val >= 0) {
            // the rays intersect above the water surface
            did_bathy = false;
            return uncorr_tri_pt;
          }
          
          for (size_t it = 0; it < 2; it++) {
            // Bend each ray at the surface according to Snell's law.
            bool ans = snells_law_curved(camCtrs[it], camDirs[it],
                                         m_bathy_set[it].bathy_plane,  
                                         m_bathy_set[it].water_surface_projection,
                                         m_refraction_index,
                                         waterCtrs[it], waterDirs[it]);
            if (!ans) {
              did_bathy = false;
              return uncorr_tri_pt;
            }
          }
        }
        
        // Re-triangulate with the new rays
        Vector3 corr_tri_pt = triangulate_point(waterDirs, waterCtrs, errorVec);
        
        did_bathy = true;
        return corr_tri_pt;
      }

      // The case of left and right images having their own bathy planes
      
      // Bend the rays
      if (!use_curved_water_surface) {
        for (size_t it = 0; it < 2; it++) {
          bool ans = snells_law(camCtrs[it], camDirs[it],
                                m_bathy_set[it].bathy_plane,  
                                m_refraction_index,
                                waterCtrs[it], waterDirs[it]);
          if (!ans)
            return uncorr_tri_pt;
        }
      } else {
        for (size_t it = 0; it < 2; it++) {
          // Bend each ray at the surface according to Snell's law.
          bool ans = snells_law_curved(camCtrs[it], camDirs[it],
                                       m_bathy_set[it].bathy_plane,  
                                       m_bathy_set[it].water_surface_projection,
                                       m_refraction_index,
                                       waterCtrs[it], waterDirs[it]);
          if (!ans)
            return uncorr_tri_pt;
        }
      }
      
      // Each ray has two parts: before bending and after it. Two
      // bent rays can intersect on their unbent parts, the bent part
      // of one ray with unbent part of another ray, unbent part of
      // one ray with bent part of another ray, and bent parts of both
      // rays. Handle all these with much care. 

      Vector3 err, tri_pt;
      std::vector<double> signed_dists;

      // See if the unbent portions intersect above their planes
      tri_pt = triangulate_pair(camDirs[0], camCtrs[0], camDirs[1], camCtrs[1], err);
      signed_distances_to_planes(use_curved_water_surface, m_bathy_set, tri_pt, signed_dists);
      if (signed_dists[0] >= 0 && signed_dists[1] >= 0) {
        did_bathy = false; // since the rays did not reach the bathy plane
        errorVec = err;
        return tri_pt;
      }
      
      // See if the bent portions intersect below their planes
      tri_pt = triangulate_pair(waterDirs[0], waterCtrs[0], waterDirs[1], waterCtrs[1], err);
      signed_distances_to_planes(use_curved_water_surface, m_bathy_set, tri_pt, signed_dists);
      if (signed_dists[0] <= 0 && signed_dists[1] <= 0) {
        did_bathy = true; // the resulting point is at least under one plane
        errorVec = err;
        return tri_pt;
      }

      // See if the left unbent portion intersects the right bent portion,
      // above left's water plane and below right's water plane
      tri_pt = triangulate_pair(camDirs[0], camCtrs[0], waterDirs[1], waterCtrs[1], err);
      signed_distances_to_planes(use_curved_water_surface, m_bathy_set, tri_pt, signed_dists);
      if (signed_dists[0] >= 0 && signed_dists[1] <= 0) {
        did_bathy = true; // the resulting point is at least under one plane
        errorVec = err;
        return tri_pt;
      }
      
      // See if the left bent portion intersects the right unbent portion,
      // below left's water plane and above right's water plane
      tri_pt = triangulate_pair(waterDirs[0], waterCtrs[0], camDirs[1], camCtrs[1], err);
      signed_distances_to_planes(use_curved_water_surface, m_bathy_set, tri_pt, signed_dists);
      if (signed_dists[0] <= 0 && signed_dists[1] >= 0) {
        did_bathy = true; // the resulting point is at least under one plane
        errorVec = err;
        return tri_pt;
      }
      
    } catch (const camera::PixelToRayErr& /*e*/) {}

    // We arrive here only when there's bad luck
    did_bathy = false;
    errorVec = vw::Vector3();
    return vw::Vector3();
  }

  Vector3 BathyStereoModel::operator()(std::vector<Vector2> const& pixVec,
                                       double& error) const {
    vw::vw_throw(vw::NoImplErr() << "Not implemented for BathyStereoModel.");
    return Vector3();
  }
  
  Vector3 BathyStereoModel::operator()(Vector2 const& pix1,
                                       Vector2 const& pix2, Vector3& errorVec) const {
    vw::vw_throw(vw::NoImplErr() << "Not implemented for BathyStereoModel.");
    return Vector3();
  }


  Vector3 BathyStereoModel::operator()(Vector2 const& pix1, Vector2 const& pix2,
                                       double& error) const {
    vw::vw_throw(vw::NoImplErr() << "Not implemented for BathyStereoModel.");
    return Vector3();
  }
  
}
