#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file DEMDisparity.cc
///

#include <vw/Image/ImageView.h>
#include <vw/Image/Transform.h>
#include <vw/Image/MaskViews.h>
#include <vw/FileIO/DiskImageView.h>
#include <vw/Camera/CameraModel.h>
#include <vw/Cartography/CameraBBox.h>
#include <vw/Cartography/GeoReferenceUtils.h>
#include <vw/Stereo/DisparityMap.h>
#include <vw/FileIO/MatrixIO.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Core/DemDisparity.h>

#include <boost/filesystem/operations.hpp>
namespace fs = boost::filesystem;

using namespace vw;
using namespace vw::cartography;

namespace asp {

  template <class ImageT, class DEMImageT>
  class DemDisparity : public ImageViewBase<DemDisparity<ImageT, DEMImageT> > {
    ImageT            m_left_image;
    double            m_dem_error;
    GeoReference      m_dem_georef;
    const DEMImageT & m_dem;
    Vector2f          m_downsample_scale;
    boost::shared_ptr<camera::CameraModel> m_left_camera_model;
    boost::shared_ptr<camera::CameraModel> m_right_camera_model;
    bool            m_do_align;
    Matrix<double>  m_align_left_matrix, m_align_right_matrix;
    int             m_pixel_sample;
    ImageView<PixelMask<Vector2i> > & m_disparity_spread;

  public:
    DemDisparity( ImageViewBase<ImageT> const& left_image,
                  double dem_error, GeoReference dem_georef,
                  DEMImageT const& dem,
                  Vector2f const& downsample_scale,
                  boost::shared_ptr<camera::CameraModel> left_camera_model,
                  boost::shared_ptr<camera::CameraModel> right_camera_model,
                  bool do_align,
                  Matrix<double> const& align_left_matrix, Matrix<double> const& align_right_matrix,
                  int pixel_sample, ImageView<PixelMask<Vector2i> > & disparity_spread)
      :m_left_image(left_image.impl()),
       m_dem_error(dem_error),
       m_dem_georef(dem_georef),
       m_dem(dem),
       m_downsample_scale(downsample_scale),
       m_left_camera_model(left_camera_model),
       m_right_camera_model(right_camera_model),
       m_do_align(do_align),
       m_align_left_matrix(align_left_matrix),
       m_align_right_matrix(align_right_matrix),
       m_pixel_sample(pixel_sample),
       m_disparity_spread(disparity_spread){}

    // Image View interface
    typedef PixelMask<Vector2i> pixel_type;
    typedef pixel_type result_type;
    typedef ProceduralPixelAccessor<DemDisparity> pixel_accessor;

    inline int32 cols  () const { return m_left_image.cols(); }
    inline int32 rows  () const { return m_left_image.rows(); }
    inline int32 planes() const { return 1; }

    inline pixel_accessor origin() const { return pixel_accessor( *this, 0, 0 ); }

    inline pixel_type operator()( double /*i*/, double /*j*/, int32 /*p*/ = 0 ) const {
      vw_throw(NoImplErr() << "DemDisparity::operator()(...) is not implemented");
      return pixel_type();
    }

    typedef CropView<ImageView<pixel_type> > prerasterize_type;
    inline prerasterize_type prerasterize(BBox2i const& bbox) const {

      prerasterize_type lowres_disparity = prerasterize_type(ImageView<pixel_type>(bbox.width(),
                                                                                   bbox.height()),
                                                             -bbox.min().x(), -bbox.min().y(),
                                                             cols(), rows() );

      for (int row = bbox.min().y(); row < bbox.max().y(); row++){
        for (int col = bbox.min().x(); col < bbox.max().x(); col++){
          lowres_disparity(col, row).invalidate();
          m_disparity_spread(col, row).invalidate();
        }
      }

      double height_error_tol = std::max(m_dem_error/4.0, 1.0); // height error in meters
      double max_abs_tol      = height_error_tol/4.0; // abs cost function change b/w iterations
      double max_rel_tol      = 1e-14;                // rel cost function change b/w iterations
      int    num_max_iter     = 50;
      bool   treat_nodata_as_zero = false;

      Vector3 prev_xyz;

      // Estimate the DEM region we expect to use and crop it into an
      // ImageView.  This will make the algorithm much faster than
      // accessing individual DEM pixels from disk. To do that, find
      // the pixel values on a small set of points of the diagonals of
      // the current tile.

      std::vector<Vector2> diagonals;
      int wid = bbox.width() - 1, hgt = bbox.height() - 1, dim = std::max(1, std::max(wid, hgt)/10);
      for (int i = 0; i <= dim; i++)
        diagonals.push_back(bbox.min() + Vector2(double(i)*wid/dim, double(i)*hgt/dim));
      for (int i = 0; i <= dim; i++)
        diagonals.push_back(bbox.min() + Vector2(double(i)*wid/dim, hgt - double(i)*hgt/dim));

      BBox2i dem_box;
      for (unsigned k = 0; k < diagonals.size(); k++){

        Vector2 left_lowres_pix = diagonals[k];

        Vector2 left_fullres_pix = elem_quot(left_lowres_pix, m_downsample_scale);
        if (m_do_align){
          // Need to go to the image pixel in the untransformed image
          left_fullres_pix = HomographyTransform(m_align_left_matrix).reverse(left_fullres_pix);
        }

        bool has_intersection;
        Vector3 left_camera_ctr, left_camera_vec;
        try {
          left_camera_ctr = m_left_camera_model->camera_center(left_fullres_pix);
          left_camera_vec = m_left_camera_model->pixel_to_vector(left_fullres_pix);
        } catch (...) {
          continue;
        }
        Vector3 xyz = camera_pixel_to_dem_xyz(left_camera_ctr, left_camera_vec,
                                              m_dem, m_dem_georef,
                                              treat_nodata_as_zero,
                                              has_intersection,
                                              height_error_tol, max_abs_tol,
                                              max_rel_tol, num_max_iter,
                                              prev_xyz);
        if ( !has_intersection || xyz == Vector3() ) continue;
        prev_xyz = xyz;

        Vector3 llh = m_dem_georef.datum().cartesian_to_geodetic( xyz );
        Vector2 pix = round(m_dem_georef.lonlat_to_pixel(subvector(llh, 0, 2)));
        dem_box.grow(pix);
      }

      // Expand the DEM box just in case as the above calculation is
      // not fool-proof if the DEM has a lot of no-data regions.
      int expand = std::max(100, (int)(0.1*std::max(dem_box.width(), dem_box.height())));
      dem_box.expand(expand);
      dem_box.crop(bounding_box(m_dem));

      // Crop the georef, read the DEM region in memory
      GeoReference georef_crop = crop(m_dem_georef, dem_box);
      ImageView <PixelMask<float> > dem_crop = crop(m_dem, dem_box);

      // Compute the DEM disparity. Use one in every 'm_pixel_sample' pixels.

      for (int row = bbox.min().y(); row < bbox.max().y(); row++){
        if (row%m_pixel_sample != 0) continue;

        // Must wipe the previous guess since we are now too far from it
        prev_xyz = Vector3();

        for (int col = bbox.min().x(); col < bbox.max().x(); col++){
          if (col%m_pixel_sample != 0) continue;

          Vector2 left_lowres_pix = Vector2(col, row);

          Vector2 left_fullres_pix = elem_quot(left_lowres_pix, m_downsample_scale);
          if (m_do_align){
            // Need to go to the image pixel in the untransformed image
            left_fullres_pix = HomographyTransform(m_align_left_matrix).reverse(left_fullres_pix);
          }

          bool has_intersection;
          Vector3 left_camera_ctr, left_camera_vec;
          try {
            left_camera_ctr = m_left_camera_model->camera_center(left_fullres_pix);
            left_camera_vec = m_left_camera_model->pixel_to_vector(left_fullres_pix);
          } catch (...) {
            continue;
          }
          Vector3 xyz = camera_pixel_to_dem_xyz(left_camera_ctr, left_camera_vec,
                                                dem_crop, georef_crop,
                                                treat_nodata_as_zero,
                                                has_intersection,
                                                height_error_tol, max_abs_tol,
                                                max_rel_tol, num_max_iter,
                                                prev_xyz
                                                );
          if ( !has_intersection || xyz == Vector3() ) continue;
          prev_xyz = xyz;

          // Since our DEM is only known approximately, the true
          // intersection point of the ray coming from the left camera
          // with the DEM could be anywhere within m_dem_error from
          // xyz. Use that to get an estimate of the disparity
          // error.

          ImageView< PixelMask<Vector2> > curr_pixel_disp_range(3, 1);
          double bias[] = {-1.0, 1.0, 0.0};
          int success[] = {0, 0, 0};

          for (int k = 0; k < curr_pixel_disp_range.cols(); k++){

            Vector2 right_fullres_pix;
            try {
              right_fullres_pix = m_right_camera_model->point_to_pixel(xyz + bias[k]*m_dem_error*left_camera_vec);
            } catch (...) {
              curr_pixel_disp_range(k, 0).invalidate();
              continue;
            }
            if (m_do_align){
              right_fullres_pix = HomographyTransform(m_align_right_matrix).forward(right_fullres_pix);
            }

            Vector2 right_lowres_pix = elem_prod(right_fullres_pix, m_downsample_scale);
            curr_pixel_disp_range(k, 0) = right_lowres_pix - left_lowres_pix;
            success[k] = 1;

            // If the disparities at the endpoints of the range were successful,
            // don't bother with the middle estimate.
            if (k == 1 && success[0] && success[1]) break;
          }

          BBox2f search_range = stereo::get_disparity_range(curr_pixel_disp_range);
          if (search_range ==  BBox2f(0,0,0,0)) continue;

          lowres_disparity(col, row) = round( (search_range.min() + search_range.max())/2.0 );
          m_disparity_spread(col, row) = ceil( (search_range.max() - search_range.min())/2.0 );

        }
      }

      return lowres_disparity;
    }

    template <class DestT>
    inline void rasterize(DestT const& dest, BBox2i bbox) const {
      vw::rasterize(prerasterize(bbox), dest, bbox);
    }
  };

  template <class ImageT, class DEMImageT>
  DemDisparity<ImageT, DEMImageT>
  dem_disparity( ImageViewBase<ImageT> const& left,
                 double dem_error, GeoReference dem_georef,
                 DEMImageT const& dem,
                 Vector2f const& downsample_scale,
                 boost::shared_ptr<camera::CameraModel> left_camera_model,
                 boost::shared_ptr<camera::CameraModel> right_camera_model,
                 bool do_align,
                 Matrix<double> const& align_left_matrix,
                 Matrix<double> const& align_right_matrix,
                 int pixel_sample,
                 ImageView<PixelMask<Vector2i> > & disparity_spread
                 ) {
    typedef DemDisparity<ImageT, DEMImageT> return_type;
    return return_type( left.impl(),
                        dem_error, dem_georef,
                        dem, downsample_scale,
                        left_camera_model, right_camera_model,
                        do_align, align_left_matrix, align_right_matrix,
                        pixel_sample, disparity_spread
                        );
  }

  void produce_dem_disparity( ASPGlobalOptions & opt,
                              boost::shared_ptr<camera::CameraModel> left_camera_model,
                              boost::shared_ptr<camera::CameraModel> right_camera_model,
                              std::string session_name
                              ) {

    if (stereo_settings().is_search_defined())
      vw_out(WarningMessage) << "Computing low-resolution disparity from DEM. "
                             << "Will ignore corr-search value: "
                             << stereo_settings().search_range << ".\n";

    // Skip pixels to speed things up, particularly for ISIS and DG.
    int pixel_sample = 2;

    DiskImageView<PixelGray<float> > left_image(opt.out_prefix+"-L.tif");
    DiskImageView<PixelGray<float> > left_image_sub(opt.out_prefix+"-L_sub.tif");

    std::string dem_file = stereo_settings().disparity_estimation_dem;
    if (dem_file == ""){
      vw_throw( ArgumentErr() << "dem_disparity: No value was provided for "
                << "disparity-estimation-dem.\n" );
    }
    double dem_error = stereo_settings().disparity_estimation_dem_error;
    if (dem_error < 0.0){
      vw_throw( ArgumentErr() << "dem_disparity: Invalid value for "
                << "disparity-estimation-dem-error: " << dem_error << ".\n" );
    }

    GeoReference dem_georef;
    bool has_georef = cartography::read_georeference(dem_georef, dem_file);
    if (!has_georef)
      vw_throw( ArgumentErr() << "There is no georeference information in: " << dem_file << ".\n" );

    DiskImageView<float> dem_disk_image(dem_file);
    ImageViewRef<PixelMask<float > > dem = pixel_cast<PixelMask<float> >(dem_disk_image);
    boost::shared_ptr<DiskImageResource> rsrc( DiskImageResourcePtr(dem_file) );
    if ( rsrc->has_nodata_read() ){
      double nodata_value = rsrc->nodata_read();
      if ( !std::isnan(nodata_value) )
        dem = create_mask(dem_disk_image, nodata_value);
    }

    Vector2f downsample_scale( float(left_image_sub.cols()) / float(left_image.cols()),
                               float(left_image_sub.rows()) / float(left_image.rows()));

    Matrix<double> align_left_matrix  = math::identity_matrix<3>();
    Matrix<double> align_right_matrix = math::identity_matrix<3>();
    bool do_align = (stereo_settings().alignment_method == "homography"     ||
                     stereo_settings().alignment_method == "affineepipolar" ||
                     stereo_settings().alignment_method == "local_epipolar");
    if (do_align){
      // We used a transform to align the images, so we have to make
      // sure to apply that transform to the disparity we are about to
      // compute as well.
      if (fs::exists(opt.out_prefix+"-align-L.exr"))
        read_matrix(align_left_matrix, opt.out_prefix + "-align-L.exr");
      if (fs::exists(opt.out_prefix+"-align-R.exr"))
        read_matrix(align_right_matrix, opt.out_prefix + "-align-R.exr");
      
      vw_out(DebugMessage,"asp") << "Left alignment matrix: "  << align_left_matrix  << "\n";
      vw_out(DebugMessage,"asp") << "Right alignment matrix: " << align_right_matrix << "\n";
    }

    // Smaller tiles is better
    Vector2 orig_tile_size = opt.raster_tile_size;
    opt.raster_tile_size = Vector2i(64, 64);

    // This image is small enough that we can keep it in memory
    ImageView<PixelMask<Vector2i> > disparity_spread(left_image_sub.cols(), left_image_sub.rows());

    ImageViewRef<PixelMask<Vector2f> > lowres_disparity
      = pixel_cast<PixelMask<Vector2f> >(dem_disparity(left_image_sub,
                                                       dem_error, dem_georef,
                                                       dem, downsample_scale,
                                                       left_camera_model, right_camera_model,
                                                       do_align,
                                                       align_left_matrix, align_right_matrix,
                                                       pixel_sample, disparity_spread
                                                       ));
    std::string disparity_file = opt.out_prefix + "-D_sub.tif";
    vw_out() << "Writing low-resolution disparity: " << disparity_file << "\n";
    if ( session_name == "isis" ){
      // ISIS does not support multi-threading
      boost::scoped_ptr<DiskImageResource> drsrc(vw::cartography::build_gdal_rsrc( disparity_file,
                                                                        lowres_disparity, opt));
      write_image(*drsrc, lowres_disparity,
                  TerminalProgressCallback("asp", "\t--> Low-resolution disparity: "));
    }else{
      vw::cartography::block_write_gdal_image(disparity_file,
                                              lowres_disparity,
                                              opt,
                                              TerminalProgressCallback
                                              ("asp", "\t--> Low-resolution disparity:") );
    }

    std::string disp_spread_file = opt.out_prefix + "-D_sub_spread.tif";
    vw_out() << "Writing low-resolution disparity spread: " << disp_spread_file << "\n";
    vw::cartography::block_write_gdal_image(disp_spread_file,
                                            disparity_spread,
                                            opt,
                                            TerminalProgressCallback
                                            ("asp", "\t--> Low-resolution disparity spread:") );
    
    // Go back to the original tile size
    opt.raster_tile_size = orig_tile_size;

#if 0 // Debug code
    ImageView<PixelMask<Vector2i> > lowres_disparity_disk;
    read_image( lowres_disparity_disk, opt.out_prefix + "-D_sub.tif" );
    ImageView<PixelMask<Vector2i> > lowres_disparity2(lowres_disparity_disk.cols()/pixel_sample, lowres_disparity_disk.rows()/pixel_sample);
    for (int col = 0; col < lowres_disparity2.cols(); col++){
      for (int row = 0; row < lowres_disparity2.rows(); row++){
        lowres_disparity2(col, row) = lowres_disparity_disk(pixel_sample*col, pixel_sample*row);
      }
    }
    vw::cartography::block_write_gdal_image( opt.out_prefix + "-D_sub2.tif",
                                             lowres_disparity2,
                                             opt,
                                             TerminalProgressCallback
                                             ("asp", "\t--> Low-resolution disparity:") );
#endif

  }

}
