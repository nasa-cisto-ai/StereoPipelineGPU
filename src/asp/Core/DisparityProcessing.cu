#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <asp/Core/StereoSettings.h>
#include <asp/Core/Common.h>
#include <asp/Core/DisparityProcessing.h>
#include <vw/Math/Transform.h>
#include <vw/Stereo/StereoModel.h>
#include <vw/FileIO/MatrixIO.h>
#include <vw/Stereo/StereoView.h>
#include <vw/Core/Stopwatch.h>
#include <vw/Cartography/Map2CamTrans.h>
#include <vw/InterestPoint/InterestData.h>

using namespace vw;
using namespace vw::cartography;

namespace asp {
  
/// Load the D_sub file
bool load_D_sub(std::string const& d_sub_file,
                ImageViewRef<PixelMask<Vector2f>> & sub_disp) {
  if (!boost::filesystem::exists(d_sub_file))
    return false;
  
  // Check the data type of the file.
  boost::shared_ptr<DiskImageResource> rsrc(DiskImageResourcePtr(d_sub_file));
  ChannelTypeEnum disp_data_type = rsrc->channel_type();
  
  if (disp_data_type == VW_CHANNEL_INT32) // Cast the integer file to float
    sub_disp = pixel_cast<PixelMask<Vector2f>>(DiskImageView<PixelMask<Vector2i>>(d_sub_file));
  else // File on disk is float
    sub_disp = DiskImageView<PixelMask<Vector2f>>(d_sub_file);
  return true;
}

// Load the low-res disparity and the scale needed to convert it to full-res
void load_D_sub_and_scale(asp::ASPGlobalOptions                    const & opt,
                          std::string                              const & d_sub_file, 
                          vw::ImageViewRef<vw::PixelMask<vw::Vector2f>>  & sub_disp,
                          vw::Vector2                                    & upsample_scale) {
  
  DiskImageView<vw::uint8> Lmask(opt.out_prefix + "-lMask.tif"),
    Rmask(opt.out_prefix + "-rMask.tif");
  
  DiskImageView<PixelGray<float> > left_sub (opt.out_prefix+"-L_sub.tif"),
    right_sub(opt.out_prefix+"-R_sub.tif");
  
  if (!load_D_sub(d_sub_file, sub_disp)) {
    std::string msg = "Could not read " + d_sub_file + ".";
    if (stereo_settings().skip_low_res_disparity_comp)
      msg += "\nPerhaps one should disable --skip-low-res-disparity-comp.";
    vw_throw(ArgumentErr() << msg << "\n");
  }

  upsample_scale = Vector2(double(Lmask.cols()) / double(sub_disp.cols()) ,
                           double(Lmask.rows()) / double(sub_disp.rows()));
}

// Filter D_sub. All alignment methods are supported.
void filter_D_sub(ASPGlobalOptions const& opt,
                  vw::TransformPtr tx_left, vw::TransformPtr tx_right,
                  boost::shared_ptr<vw::camera::CameraModel> left_camera_model, 
                  boost::shared_ptr<vw::camera::CameraModel> right_camera_model,
                  vw::cartography::Datum const& datum,
                  std::string const& d_sub_file,
                  Vector2 const& outlier_removal_params) {
  
  if (outlier_removal_params[0] >= 100.0)
    return; // The user chose to skip outlier filtering
  
  vw_out() << "Filtering outliers in D_sub based on --outlier-removal-params.\n";

  double pct = outlier_removal_params[0];
  double factor = outlier_removal_params[1];
  double pct_fraction = 1.0 - pct/100.0;

  vw::ImageViewRef<vw::PixelMask<vw::Vector2f>> sub_disp_ref;
  vw::Vector2 upsample_scale;
  asp::load_D_sub_and_scale(opt, d_sub_file, sub_disp_ref, upsample_scale);

  // Use ImageView to read D_sub fully in memory so it can be modified
  vw::ImageView<vw::PixelMask<vw::Vector2f>> sub_disp = sub_disp_ref;

  // Will save the sub PC file, since we compute these anyway
  // for the purpose of filtering.
  vw::ImageView<vw::Vector<double, 4>> sub_pc(sub_disp.cols(), sub_disp.rows());
  for (int col = 0; col < sub_pc.cols(); col++) {
    for (int row = 0; row < sub_pc.rows(); row++) {
      for (int coord = 0; coord < 4; coord++) {
        sub_pc(col, row)[coord] = 0;
      }
    }
  }
  
  // Find the disparity values in x and y
  std::vector<double> dx, dy;
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      vw::PixelMask<vw::Vector2f> disp = sub_disp(col, row);
      
      if (!is_valid(disp)) 
        continue;
      
      Vector2 left_pix(col, row);
      Vector2 right_pix = left_pix + disp.child();

      double diffX = right_pix.x() - left_pix.x();
      double diffY = right_pix.y() - left_pix.y();
      dx.push_back(diffX);
      dy.push_back(diffY);
    }
  }
  
  if (dx.empty())
    vw_throw(ArgumentErr() << "Empty disparity.");

  // Find the outlier brackets based on values in x and y
  double bx = -1.0, ex = -1.0;
  vw::math::find_outlier_brackets(dx, pct_fraction, factor, bx, ex);
  vw_out() <<"Inlier range based on x coordinate of disparity: " << bx << ' ' << ex <<".\n";

  double by = -1.0, ey = -1.0;
  vw::math::find_outlier_brackets(dy, pct_fraction, factor, by, ey);
  vw_out() <<"Inlier range based on y coordinate of disparity: " << by << ' ' << ey <<".\n";

  int count = 0;
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      vw::PixelMask<vw::Vector2f> disp = sub_disp(col, row);
      
      if (!is_valid(disp)) 
        continue;
      
      Vector2 left_pix(col, row);
      Vector2 right_pix = left_pix + disp.child();

      double diffX = right_pix.x() - left_pix.x();
      double diffY = right_pix.y() - left_pix.y();

      if (diffX < bx || diffX > ex || diffY < by || diffY > ey) {
        sub_disp(col, row).invalidate();
        count++;
      }
    }
  }
  vw_out() << "Number (and fraction) of removed outliers by disparity values in x and y: "
           << count << " (" << double(count)/(sub_disp.cols() * sub_disp.rows()) << ").\n";
  
  // Set up the stereo model for doing triangulation
  double angle_tol = vw::stereo::StereoModel
    ::robust_1_minus_cos(stereo_settings().min_triangulation_angle*M_PI/180);
  stereo::StereoModel model(left_camera_model.get(), right_camera_model.get(),
                            stereo_settings().use_least_squares, angle_tol);

  float HIGH_ERROR = std::numeric_limits<float>::max();
  ImageView<float> tri_err(sub_disp.cols(), sub_disp.rows());
  ImageView<float> height(sub_disp.cols(), sub_disp.rows());

  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      vw::PixelMask<vw::Vector2f> disp = sub_disp(col, row);
      
      if (!is_valid(disp)) {
        tri_err(col, row) = HIGH_ERROR;
        height(col, row) = HIGH_ERROR;
        continue;
      }
      
      Vector2 left_pix(col, row);
      Vector2 right_pix = left_pix + disp.child();

      // Scale to full resolution
      left_pix  = elem_prod(left_pix, upsample_scale);
      right_pix = elem_prod(right_pix, upsample_scale);

      // Undo the alignment transform
      left_pix = tx_left->reverse(left_pix);
      right_pix = tx_right->reverse(right_pix);

      double err;
      Vector3 xyz;
      try {
        xyz = model(left_pix, right_pix, err);
      } catch(...) {
        xyz = Vector3();
      }
      
      // The call returns the zero error and zero xyz to indicate a
      // failed ray intersection so replace it in those cases with a
      // very high error.
      if (err == 0 || xyz == Vector3()) {
        tri_err(col, row) = HIGH_ERROR;
        height(col, row) = HIGH_ERROR;
        continue;
      }
        
      tri_err(col, row) = err;

      // Save the triangulated point
      Vector<double, 4> P;
      subvector(P, 0, 3) = xyz;
      P[3] = err;
      sub_pc(col, row) = P;
      
      Vector3 llh = datum.cartesian_to_geodetic(xyz);
      height(col, row) = llh[2];
    }
  }

  // Put the valid heights in a vector
  std::vector<double> vals;
  // Careful below to avoid integer overflow
  vals.reserve(std::int64_t(sub_disp.cols()) * std::int64_t(sub_disp.rows()));
  vals.clear();
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      if (height(col, row) >= HIGH_ERROR) continue; // already invalid
      vals.push_back(height(col, row));
    }
  }

  // Find the outlier brackets
  double b = -1.0, e = -1.0;
  vw::math::find_outlier_brackets(vals, pct_fraction, factor, b, e);
  vw_out() <<"Height above datum inlier range: " << b << ' ' << e <<".\n";

  // Apply the outlier threshold
  count = 0;
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      if (height(col, row) >= HIGH_ERROR) continue; // already invalid
      if (height(col, row) < b || height(col, row) > e) {
        height(col, row)  = HIGH_ERROR;
        tri_err(col, row) = HIGH_ERROR;
        count++;
      }
    }
  }
  vw_out() << "Number (and fraction) of removed outliers by the height check: "
           << count << " (" << double(count)/(sub_disp.cols() * sub_disp.rows()) << ").\n";
    
  // Put the tri errors in a vector
  vals.clear();
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      if (tri_err(col, row) >= HIGH_ERROR) continue; // already invalid
      vals.push_back(tri_err(col, row));
    }
  }

  // Find the outlier brackets. Since the triangulation errors, unlike
  // the heights, are usually rather uniform, adjust pct from 95 to
  // 90.
  double pct2 = std::max((90.0/95.0) * outlier_removal_params[0], 0.5);
  double pct_fraction2 = 1.0 - pct2/100.0;
  // Show some lenience below as due to jitter some errors could be somewhat bigger
  double factor2 = 2.0 * factor;
  b = -1.0;
  e = -1.0;
  vw::math::find_outlier_brackets(vals, pct_fraction2, factor2, b, e);
  vw_out() <<"Triangulation error inlier range: " << b << ' ' << e <<".\n";
    
  // Apply the outlier threshold
  count = 0;
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      if (tri_err(col, row) >= HIGH_ERROR) continue; // already invalid
      // We will ignore b, as the triangulation errors are non-negative.
      if (tri_err(col, row) > e) {
        height(col, row) = HIGH_ERROR;
        tri_err(col, row) = HIGH_ERROR;
        count++;
      }
    }
  }
  vw_out() << "Number (and fraction) of removed outliers by the triangulation error check: "
           << count << " (" << double(count)/(sub_disp.cols() * sub_disp.rows()) << ").\n";

  // TODO(oalexan1): Filter by user-given height range and max tri error.
    
  // Invalidate the D_sub entries that are outliers
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      if (tri_err(col, row) >= HIGH_ERROR || height(col, row) >= HIGH_ERROR) {
        sub_disp(col, row).invalidate();

        // Invalidate the point in the cloud
        subvector(sub_pc(col, row), 0, 3) = Vector3(0.0, 0.0, 0.0);
        sub_pc(col, row)[3] = 0.0;
      }
    }
  }

  vw_out() << "Writing filtered D_sub: " << d_sub_file << std::endl;
  block_write_gdal_image(d_sub_file, sub_disp, opt,
                         TerminalProgressCallback("asp","\t D_sub: "));

  std::string pc_sub_file = opt.out_prefix + "-PC_sub.tif";
  vw_out() << "Writing triangulated point cloud based on D_sub: " << pc_sub_file << std::endl;
  block_write_gdal_image(pc_sub_file, sub_pc, opt,
                         TerminalProgressCallback("asp","\t PC_sub: "));
  
} 

// Filter D_sub by reducing its spread around the median
void filter_D_sub_using_spread(ASPGlobalOptions const& opt, std::string const& d_sub_file,
                               double max_disp_spread) {
  
  if (max_disp_spread <= 0.0)
    return;

  vw_out() << "Filtering outliers in D_sub based on --max-disp-spread.\n";

  vw::ImageViewRef<vw::PixelMask<vw::Vector2f>> sub_disp_ref;
  vw::Vector2 upsample_scale;
  asp::load_D_sub_and_scale(opt, d_sub_file, sub_disp_ref, upsample_scale);

  // Use ImageView to read D_sub fully in memory so it can be modified
  vw::ImageView<vw::PixelMask<vw::Vector2f>> sub_disp = sub_disp_ref;

  std::vector<double> dx, dy;
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      vw::PixelMask<vw::Vector2f> disp = sub_disp(col, row);
      
      if (!is_valid(disp)) 
        continue;
      
      Vector2 left_pix(col, row);
      Vector2 right_pix = left_pix + disp.child();

      // Scale to full resolution
      left_pix  = elem_prod(left_pix, upsample_scale);
      right_pix = elem_prod(right_pix, upsample_scale);

      double diffX = right_pix.x() - left_pix.x();
      double diffY = right_pix.y() - left_pix.y();
      dx.push_back(diffX);
      dy.push_back(diffY);
    }
  }
  
  if (dx.empty())
    vw_throw(ArgumentErr() << "Empty disparity.");
  
  std::sort(dx.begin(), dx.end());
  std::sort(dy.begin(), dy.end());
  double mid_x = dx[dx.size()/2]; // median
  double mid_y = dy[dy.size()/2];
  
  double half = max_disp_spread / 2.0;
  BBox2 spread_box(mid_x - half, mid_y - half, max_disp_spread, max_disp_spread);

  // Wipe offending disparities
  int count = 0;
  for (int col = 0; col < sub_disp.cols(); col++) {
    for (int row = 0; row < sub_disp.rows(); row++) {
      vw::PixelMask<vw::Vector2f> disp = sub_disp(col, row);
      
      if (!is_valid(disp)) 
        continue;
      
      Vector2 left_pix(col, row);
      Vector2 right_pix = left_pix + disp.child();

      // Scale to full resolution
      left_pix  = elem_prod(left_pix, upsample_scale);
      right_pix = elem_prod(right_pix, upsample_scale);

      double diffX = right_pix.x() - left_pix.x();
      double diffY = right_pix.y() - left_pix.y();

      Vector2 d(diffX, diffY);
      if (!spread_box.contains(d)) {
        count++;
        sub_disp(col, row).invalidate();
      }
    }
  }

  vw_out() << "Number (and fraction) of removed outliers by the disp spread check: "
           << count << " (" << double(count)/(sub_disp.cols() * sub_disp.rows()) << ").\n";
    
  vw_out() << "Writing filtered D_sub: " << d_sub_file << std::endl;
  block_write_gdal_image(d_sub_file, sub_disp, opt,
                         TerminalProgressCallback("asp","\t D_sub: "));
} 

// Compute an unaligned disparity image from the input disparity image
// and the image transforms.
// Note that the output image size is not the same as the input disparity image.
class UnalignDisparityView: public ImageViewBase<UnalignDisparityView>{
  
  DispImageType    const& m_disparity;
  vw::TransformPtr const& m_left_transform;
  vw::TransformPtr const& m_right_transform;

  ASPGlobalOptions const& m_opt;
  int m_num_cols, m_num_rows;
  bool m_is_map_projected;
  std::map <std::pair<int, int>, Vector2> m_unaligned_trans;
public:
  UnalignDisparityView(bool is_map_projected,
                       DispImageType    const& disparity,
                       vw::TransformPtr const& left_transform,
                       vw::TransformPtr const& right_transform,
                       ASPGlobalOptions const& opt):
    m_is_map_projected(is_map_projected), 
    m_disparity(disparity), m_left_transform(left_transform), 
    m_right_transform(right_transform), m_opt(opt),
    m_num_cols(0), m_num_rows(0){

    // Compute the output image size
    
    if (!m_is_map_projected) {
      // The left image passed as input is the original
      // unprojected/unaligned one, hence use its size.
      std::string left_file  = m_opt.in_file1;
      DiskImageView<float> left_img(left_file);
      m_num_cols = left_img.cols();
      m_num_rows = left_img.rows();
    }else{
      // Map projected, need to check all the pixel coordinates.
      // This is going to be slow for large images!
      BBox2i img_box;

      // Use sampling as this operation is very slow.
      int sample_len = 10;
      int num_min_samples = 100;
      int col_sample = std::max(1, std::min(sample_len, m_disparity.cols()/num_min_samples));
      int row_sample = std::max(1, std::min(sample_len, m_disparity.rows()/num_min_samples));

      vw::TerminalProgressCallback tpc("asp", "\t--> ");
      double inc_amount = col_sample / std::max(double(m_disparity.cols()), 1.0);
      tpc.report_progress(0);
      vw_out() << "\nEstimating the unaligned disparity dimensions.\n";

      for (int col = 0; col < m_disparity.cols(); col++) {

	// Ensure that the last column is picked
	if (col % col_sample != 0 && col != m_disparity.cols() - 1) 
	  continue;

        for (int row = 0; row < m_disparity.rows(); row++) {
	  
	  // Ensure that the last row is picked
	  if (row % row_sample != 0 && row != m_disparity.rows() - 1) 
	    continue;

	  // This is quite important to avoid an incorrectly computed img_box.
          typename DispImageType::pixel_type dpix = m_disparity(col, row);
          if (!is_valid(dpix))
	    continue;

          // Unalign the left pixel
	  Vector2 left_pix;
	  try{
	    left_pix  = m_left_transform->reverse(Vector2(col, row));
	  }catch(...){
	    continue;
	  }
          img_box.grow(left_pix);

	  // Save this lookup map for the future
	  m_unaligned_trans[std::make_pair(col, row)] = left_pix;
        }

	tpc.report_incremental_progress(inc_amount);
      }
      tpc.report_finished();

      // Grow the box to account for the fact that we did a sub-sampling
      // and may have missed some points.
      Vector2 diff = img_box.max() - img_box.min();
      if (!img_box.empty()) {
	img_box.grow(img_box.min() - 0.1*diff);
	img_box.grow(img_box.max() + 0.1*diff);
      }
      
      m_num_cols = img_box.max().x();
      m_num_rows = img_box.max().y();

      vw_out() << "Dimensions are: " << m_num_cols << ' ' << m_num_rows << ".\n";
    }
    // Done computing the input image size.
  }

  // ImageView interface
  typedef PixelMask<Vector2f>                          pixel_type;
  typedef pixel_type                                   result_type;
  typedef ProceduralPixelAccessor<UnalignDisparityView> pixel_accessor;

  inline int32 cols  () const { return m_num_cols; }
  inline int32 rows  () const { return m_num_rows; }
  inline int32 planes() const { return 1; }

  inline pixel_accessor origin() const { return pixel_accessor(*this, 0, 0); }

  inline pixel_type operator()(double /*i*/, double /*j*/, int32 /*p*/ = 0) const {
    vw_throw(vw::NoImplErr() << "UnalignDisparityView::operator()(...) is not implemented");
    return pixel_type();
  }

  typedef CropView<ImageView<pixel_type> > prerasterize_type;
  inline prerasterize_type prerasterize(BBox2i const& bbox) const {

    vw::TransformPtr local_left_transform;
    vw::TransformPtr local_right_transform;

    // For map-projected images the transforms are not thread-safe,
    // hence need to make a copy of them.
    if (!m_is_map_projected) {
      local_left_transform = m_left_transform;
      local_right_transform = m_right_transform;
    }else{
      local_left_transform = vw::cartography::mapproj_trans_copy(m_left_transform);
      local_right_transform = vw::cartography::mapproj_trans_copy(m_right_transform);
    }

    // We will do some averaging
    int KERNEL_SIZE = 1;
    
    BBox2i curr_bbox = bbox;
    curr_bbox.expand(2*KERNEL_SIZE);
    curr_bbox.crop(BBox2i(0, 0, cols(), rows()));
    
    // Initialize the unaligned disparity values for this tile.
    ImageView<pixel_type> unaligned_disp(curr_bbox.width(), curr_bbox.height());
    ImageView<int> count(curr_bbox.width(), curr_bbox.height());
    for (int col = 0; col < curr_bbox.width(); col++) {
      for (int row = 0; row < curr_bbox.height(); row++) {
        unaligned_disp(col, row) = pixel_type();
        unaligned_disp(col, row).invalidate();
	count(col, row) = 0;
      }
    }
    
    // Find the bounding box of pixels we will need from the disparity image.
    // For mapprojected images the forward() function is not always accurate,
    // and it is also very slow, hence avoid it.
    BBox2i disp_bbox;
    if (!m_is_map_projected) {
      BBox2i full_disp_bbox = bounding_box(m_disparity);
      for (int col = 0; col < unaligned_disp.cols(); col++) {
	for (int row = 0; row < unaligned_disp.rows(); row++) {
	  
	  // Get the pixel coordinate in the output image (left unaligned pixel),
	  // Then get the pixel coordinate in the left input image.
	  Vector2 output_pixel(col + curr_bbox.min()[0], row + curr_bbox.min()[1]);
	  Vector2 left_aligned_pixel;
	try {
	  left_aligned_pixel = local_left_transform->forward(output_pixel);
	}catch(...){
	  // This can fail since we may apply it to pixels outside of range
	  continue;
	}
	if (!full_disp_bbox.contains(left_aligned_pixel)) 
	  continue;
	disp_bbox.grow(left_aligned_pixel);
	}
      }
    }else{
      for (int col = 0; col < m_disparity.cols(); col++) {
	for (int row = 0; row < m_disparity.rows(); row++) {
	  
	  std::pair<int, int> pix = std::make_pair(col, row);
	  std::map <std::pair<int, int>, Vector2>::const_iterator it = m_unaligned_trans.find(pix);
	  if (it == m_unaligned_trans.end())
	    continue;

	  Vector2 rev = it->second;
	  if (curr_bbox.contains(rev)) {
	    disp_bbox.grow(Vector2(col, row));
	  }
	}
      }

      // Grow the box to account for the fact that we did a sub-sampling
      // and may have missed some points.
      Vector2 diff = disp_bbox.max() - disp_bbox.min();
      if (!disp_bbox.empty()) {
	disp_bbox.grow(disp_bbox.min() - 0.1*diff);
	disp_bbox.grow(disp_bbox.max() + 0.1*diff);
      }
      
    }
    
    // Expand to take into account the sampling to be used below
    disp_bbox.expand(2*KERNEL_SIZE);

    // Crop to its maximum extent
    disp_bbox.crop(bounding_box(m_disparity));

    // Rasterize the section of the disparity image that we need for this tile
    typedef typename DispImageType::pixel_type DispPixelT;
    ImageView<DispPixelT> disp = crop(m_disparity, disp_bbox);

    for (int col = 0; col < disp.cols(); col++) {
      for (int row = 0; row < disp.rows(); row++) {
	
	DispPixelT dpix = disp(col, row);
	if (!is_valid(dpix))
	  continue;

	// Go from position in the cropped disparity to the
	// position in the full disparity.
	int ucol = col + disp_bbox.min().x();
	int urow = row + disp_bbox.min().y();
	
	// De-warp left and right pixels to be in the camera coordinate system
	Vector2 left_pix, right_pix;
	try{
	  left_pix  = local_left_transform->reverse (Vector2(ucol, urow));
	  right_pix = local_right_transform->reverse(Vector2(ucol, urow)
                                                     + stereo::DispHelper(dpix));
	}catch(...){
	  continue;
	}
	Vector2 dir = right_pix - left_pix; // disparity value
	
	// This averaging is useful in filling tiny holes and avoiding staircasing.
	// TODO: Use some weights. The closer contribution should have more weight.
	for (int icol = -KERNEL_SIZE; icol <= KERNEL_SIZE; icol++) {
	  for (int irow = -KERNEL_SIZE; irow <= KERNEL_SIZE; irow++) {
	    int lcol = round(left_pix[0]) + icol;
	    int lrow = round(left_pix[1]) + irow;
	    
	    // shift to be in the domain of the cropped image
	    lcol -= curr_bbox.min()[0];
	    lrow -= curr_bbox.min()[1];
	    if (lcol < 0 || lcol >= curr_bbox.width())  continue;
	    if (lrow < 0 || lrow >= curr_bbox.height()) continue;
	    if (!is_valid(unaligned_disp(lcol, lrow)))
	      unaligned_disp(lcol, lrow).validate();
	    unaligned_disp(lcol, lrow).child() += dir;
	    count(lcol, lrow)++;
	  }
	}
	
      }
    }
    
    for (int col = 0; col < unaligned_disp.cols(); col++) {
      for (int row = 0; row < unaligned_disp.rows(); row++) {
	if (count(col, row) == 0)
	  unaligned_disp(col, row).invalidate();
	else
	  unaligned_disp(col, row) /= double(count(col, row));
      }
    }
    
    // Use the crop trick to fake that the support region is the same size as the entire image.
    return prerasterize_type(unaligned_disp, -curr_bbox.min().x(), -curr_bbox.min().y(),
			     cols(), rows());
  }
  
  template <class DestT>
  inline void rasterize(DestT const& dest, BBox2i bbox) const {
    vw::rasterize(prerasterize(bbox), dest, bbox);
  }
}; // End class UnalignDisparityView

// Take a given disparity and make it between the original unaligned images
void unalign_disparity(bool is_map_projected,
                       DispImageType    const& disparity,
                       vw::TransformPtr const& left_trans,
                       vw::TransformPtr const& right_trans,
                       ASPGlobalOptions const& opt,
                       std::string      const& disp_file) {
  Stopwatch sw;
  sw.start();

  cartography::GeoReference left_georef;
  bool   has_left_georef = false;
  bool   has_nodata      = false;
  double nodata          = -32768.0;
  vw_out() << "Unaligning the disparity.\n";
  vw_out() << "Writing: " << disp_file << "\n";
  vw::cartography::block_write_gdal_image
    (disp_file, 
     UnalignDisparityView(is_map_projected, disparity, left_trans, right_trans, opt),
     has_left_georef, left_georef,
     has_nodata, nodata, opt,
     TerminalProgressCallback("asp", "\t--> Undist disp:"));
  
  sw.stop();
  vw_out() << "Unaligning disparity elapsed time: " << sw.elapsed_seconds() << " seconds.\n";
  
}

/// Bin the disparities, and from each bin get a disparity value.
/// This will create a correspondence from the left to right image,
/// which we save in the match format.
/// When gen_triplets is true, and there are many overlapping images,
/// try hard to have many IP with the property that each such IP is seen
/// in more than two images. This helps with bundle adjustment.
void compute_matches_from_disp(ASPGlobalOptions const& opt,
                               DispImageType    const& disp,
                               vw::TransformPtr const& left_trans,
                               vw::TransformPtr const& right_trans,
                               std::string      const& match_file,
                               int max_num_matches,
                               bool gen_triplets, bool is_map_projected) {

  std::vector<vw::ip::InterestPoint> left_ip, right_ip;

  if (!gen_triplets) {

    // Use doubles to avoid integer overflow
    double num_pixels = double(disp.cols()) * double(disp.rows());
    double bin_len = sqrt(num_pixels/std::min(double(max_num_matches), num_pixels));
    VW_ASSERT(bin_len >= 1.0, vw::ArgumentErr() << "Expecting bin_len >= 1.\n");

    int lenx = round(disp.cols()/bin_len); lenx = std::max(1, lenx);
    int leny = round(disp.rows()/bin_len); leny = std::max(1, leny);

    // Iterate over bins.

    vw_out() << "Computing interest point matches based on disparity.\n";
    vw::TerminalProgressCallback tpc("asp", "\t--> ");
    double inc_amount = 1.0 / double(lenx);
    tpc.report_progress(0);

    for (int binx = 0; binx < lenx; binx++) {

      // Pick the disparity at the center of the bin
      int posx = round((binx+0.5)*bin_len);

      for (int biny = 0; biny < leny; biny++) {

        int posy = round((biny+0.5)*bin_len);

        if (posx >= disp.cols() || posy >= disp.rows()) 
          continue;
        typedef typename DispImageType::pixel_type DispPixelT;
        DispPixelT dpix = disp(posx, posy);
        if (!is_valid(dpix))
          continue;

        // De-warp left and right pixels to be in the camera coordinate system
        Vector2 left_pix  = left_trans->reverse (Vector2(posx, posy));
        Vector2 right_pix = right_trans->reverse(Vector2(posx, posy) + stereo::DispHelper(dpix));

        left_ip.push_back(ip::InterestPoint(left_pix.x(), left_pix.y()));
        right_ip.push_back(ip::InterestPoint(right_pix.x(), right_pix.y()));
      }

      tpc.report_incremental_progress(inc_amount);
    }
    tpc.report_finished();

  } else{

    // First create ip with left_ip being at integer multiple of bin size.
    // Then do the same for right_ip. This way there is a symmetry
    // and predictable location for ip. So if three images overlap,
    // a feature can often be seen in many of them whether a given
    // image is left in some pairs or right in some others.

    // Note that the code above is modified in subtle ways.

    // Interpolate the disparity. When interpolating out-of-range
    // pixels, return invalid values.
    typedef typename DispImageType::pixel_type DispPixelT;
    DispPixelT invalid_disp; invalid_disp.invalidate();
    vw::ValueEdgeExtension<DispPixelT> invalid_ext(invalid_disp);
    auto interp_disp = interpolate(disp, BilinearInterpolation(), invalid_ext);

    // Need these to not insert an ip twice, as then bundle_adjust
    // will wipe both copies
    std::map<double, double> left_done, right_done;
    
    // Start with the left
    {
      DiskImageView<float> left_img(opt.in_file1);

      vw::BBox2 box = vw::bounding_box(left_img);
      // If images are mapprojected, need to find the bounding box of
      // unaligned pixels.
      if (is_map_projected) {
        
        // This is very slow. For now, just turn off this logic until it is studied more.
        vw::vw_throw(vw::ArgumentErr() << "Option --num-matches-from-disp-triplets does not work with mapprojected images. Use instead --num-matches-from-disparity.\n");
        box = left_trans->reverse_bbox(box); 
      }
      
      // Use doubles to avoid integer overflow
      double num_pixels = double(left_img.cols()) * double(left_img.rows());
      int bin_len = round(sqrt(num_pixels/std::min(double(max_num_matches), num_pixels)));
      VW_ASSERT(bin_len >= 1, vw::ArgumentErr() << "Expecting bin_len >= 1.\n");

      int lenx = round(left_img.cols()/bin_len); lenx = std::max(1, lenx);
      int leny = round(left_img.rows()/bin_len); leny = std::max(1, leny);

      // Iterate over bins.

      vw_out() << "Computing interest point matches based on disparity.\n";
      vw::TerminalProgressCallback tpc("asp", "\t--> ");
      double inc_amount = 1.0 / double(lenx);
      tpc.report_progress(0);

      for (int binx = 0; binx <= lenx; binx++) {

        int posx = binx*bin_len; // integer multiple of bin length

        for (int biny = 0; biny <= leny; biny++) {

          int posy = biny*bin_len; // integer multiple of bin length

          if (posx >= left_img.cols() || posy >= left_img.rows()) 
            continue;

          Vector2 left_pix(posx, posy);
          Vector2 trans_left_pix, trans_right_pix, right_pix;
          
          // Make the left pixel go to the disparity domain. Find the corresponding
          // right pixel. And make that one go to the right image domain.
          // left_pix is unaligned left pixel, while trans_left_pix is the aligned one.
          // This one is no longer integer, so need to interpolate to find its
          // disparity.
          trans_left_pix = left_trans->forward(left_pix);
          if (trans_left_pix[0] < 0 || trans_left_pix[0] >= disp.cols()) continue;
          if (trans_left_pix[1] < 0 || trans_left_pix[1] >= disp.rows()) continue;
          DispPixelT dpix = interp_disp(trans_left_pix[0], trans_left_pix[1]);
          if (!is_valid(dpix))
            continue;
          trans_right_pix = trans_left_pix + stereo::DispHelper(dpix);
          right_pix = right_trans->reverse(trans_right_pix);

          // Add this ip unless found already. This is clumsy, but we
          // can't use a set since there is no ordering for pairs.
          std::map<double, double>::iterator it;
          it = left_done.find(left_pix.x());
          if (it != left_done.end() && it->second == left_pix.y()) continue; 
          it = right_done.find(right_pix.x());
          if (it != right_done.end() && it->second == right_pix.y()) continue; 
          left_done[left_pix.x()] = left_pix.y();
          right_done[right_pix.x()] = right_pix.y();
          ip::InterestPoint lip(left_pix.x(), left_pix.y());
          ip::InterestPoint rip(right_pix.x(), right_pix.y());
          left_ip.push_back(lip); 
          right_ip.push_back(rip);
        }
        
        tpc.report_incremental_progress(inc_amount);
      }
      tpc.report_finished();
    }
    
    // Now create ip in predictable locations for the right image. This is hard,
    // as the disparity goes from left to right, so we need to examine every disparity.
    typedef typename DispImageType::pixel_type DispPixelT;
    {
      DiskImageView<float> right_img(opt.in_file2);
    
      double num_pixels = double(right_img.cols()) * double(right_img.rows());
      int bin_len = round(sqrt(num_pixels/std::min(double(max_num_matches), num_pixels)));
      VW_ASSERT(bin_len >= 1, vw::ArgumentErr() << "Expecting bin_len >= 1.\n");

      // Iterate over disparity.

      vw_out() << "Doing a second pass. This will be very slow.\n";
      vw::TerminalProgressCallback tpc("asp", "\t--> ");
      double inc_amount = 1.0 / double(disp.cols());
      tpc.report_progress(0);

      for (int col = 0; col < disp.cols(); col++) {
        for (int row = 0; row < disp.rows(); row++) {

          Vector2 trans_left_pix(col, row);
          Vector2 left_pix, trans_right_pix, right_pix;

          DispPixelT dpix = disp(trans_left_pix[0], trans_left_pix[1]);
          if (!is_valid(dpix))
            continue;

          // Compute the left and right pixels. 
          left_pix        = left_trans->reverse(trans_left_pix);
          trans_right_pix = trans_left_pix + stereo::DispHelper(dpix);
          right_pix       = right_trans->reverse(trans_right_pix);

          // If the right pixel is a multiple of the bin size, keep
          // it.
          right_pix = round(right_pix); // very important
          if (int(right_pix[0]) % bin_len != 0) continue;
          if (int(right_pix[1]) % bin_len != 0) continue;

          // Add this ip unless found already. This is clumsy, but we
          // can't use a set since there is no ordering for pairs.
          std::map<double, double>::iterator it;
          it = left_done.find(left_pix.x());
          if (it != left_done.end() && it->second == left_pix.y()) continue; 
          it = right_done.find(right_pix.x());
          if (it != right_done.end() && it->second == right_pix.y()) continue; 
          left_done[left_pix.x()] = left_pix.y();
          right_done[right_pix.x()] = right_pix.y();
          ip::InterestPoint lip(left_pix.x(), left_pix.y());
          ip::InterestPoint rip(right_pix.x(), right_pix.y());
          left_ip.push_back(lip); 
          right_ip.push_back(rip);
        }
        
        tpc.report_incremental_progress(inc_amount);
      }
      tpc.report_finished();
    }
    
  } // end considering multi-image friendly ip

  vw_out() << "Determined " << left_ip.size()
           << " interest point matches from disparity.\n";

  vw_out() << "Writing: " << match_file << std::endl;
  ip::write_binary_match_file(match_file, left_ip, right_ip);
}
  
} // end namespace asp
