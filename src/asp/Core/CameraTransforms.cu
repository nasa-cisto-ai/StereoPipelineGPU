// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file CameraTransforms.cc

#include <asp/Core/CameraTransforms.h>

#include <Eigen/Dense>
#include <Eigen/Geometry>

namespace asp {

// A function to convert a 3x3 VW matrix to Eigen
Eigen::Matrix3d vwToEigenMat(vw::Matrix3x3 const& m) {
  Eigen::Matrix3d result;
  for (int r = 0; r < 3; r++)
    for (int c = 0; c < 3; c++)
      result(r, c) = m(r, c);
  return result;
}

// Convert an Eigen 3x3 matrix to a VW 3x3 matrix
vw::Matrix3x3 eigenToVwMat(Eigen::Matrix3d const& m) {
  vw::Matrix3x3 result;
  for (int r = 0; r < 3; r++)
    for (int c = 0; c < 3; c++)
      result(r, c) = m(r, c);
  return result;
}

// A function to go from a VW matrix to a quaternion, represented
// as four values, x, y, z, w. Care with the order of values. It is not
// w, x, y, z as in the Eigen convention.
void matrixToQuaternion(vw::Matrix3x3 const& R, 
                         // Outputs
                         double & x, double & y, double & z, double & w) {

    // Convert to Eigen3
    Eigen::Matrix3d M = vwToEigenMat(R);
    // Convert to quaternion
    Eigen::Quaterniond q(M);
    // Convert to x, y, z, w
    x = q.x(); y = q.y(); z = q.z(); w = q.w();
}

// A function to convert a quaternion given by 4 numbers to a VW matrix. It is
// very important to note that were we assume the order of the quaternion
// numbers is (x, y, z, w) and not (w, x, y, z).
vw::Matrix3x3 quaternionToMatrix(double const* q) {
  Eigen::Quaterniond eq;
  eq.x() = q[0];
  eq.y() = q[1];
  eq.z() = q[2];
  eq.w() = q[3];
  // Normalize it
  eq.normalize();
  Eigen::Matrix3d m = eq.toRotationMatrix();

  // Convert to vw::Matrix3x3
  return eigenToVwMat(m);
}

// Find the roll-pitch-yaw rotation in ZYX order. The inputs are in
// degrees.
vw::Matrix3x3 rollPitchYaw(double roll, double pitch, double yaw) {

    // Factor to convert from degrees to radians
    const double DEG_TO_RAD = M_PI/180.0;

    Eigen::AngleAxisd rollAngle (DEG_TO_RAD * roll,  Eigen::Vector3d::UnitX());
    Eigen::AngleAxisd pitchAngle(DEG_TO_RAD * pitch, Eigen::Vector3d::UnitY());
    Eigen::AngleAxisd yawAngle  (DEG_TO_RAD * yaw,   Eigen::Vector3d::UnitZ());

    // Multiply these returing an Eigen Matrix3d
    Eigen::Quaterniond q = yawAngle * pitchAngle * rollAngle;

    // Convert to Eigen3
    Eigen::Matrix3d M = q.matrix();

    // Convert to vw::Matrix3x3
    vw::Matrix3x3 R = eigenToVwMat(M);
    
    return R;
}
    
// Given a matrix obtained by multiplying roll, pitch, and yaw rotations, by applying
// then in this order from right to left, find the roll, pitch, and yaw angles in degrees.
// This can return 180 +/- x, if x was the original angle, so it is not a true inverse.
void rollPitchYawFromRotationMatrix(vw::Matrix3x3 const& R, 
  // Outputs
  double & roll, double & pitch, double & yaw) {

    // Convert to Eigen3 and then find the Euler angles in radians
    Eigen::Matrix3d M = vwToEigenMat(R);
    Eigen::Vector3d euler = M.eulerAngles(2, 1, 0);

    // Convert to degrees
    roll  = euler[2]*180.0/M_PI;
    pitch = euler[1]*180.0/M_PI;
    yaw   = euler[0]*180.0/M_PI;
}

} //end namespace asp

