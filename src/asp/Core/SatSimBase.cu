#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Very low-level functions used in sat_sim.cc and jitter_solve.cc. 

#include <asp/Core/SatSimBase.h>
#include <vw/Cartography/GeoReferenceUtils.h>

using namespace vw::cartography;

namespace asp {

// A small number to help convert directions from being in projected space to
// ECEF (the transform between these is nonlinear). Do not use a small value,
// as in ECEF these will be large numbers and we may have precision issues.
// The value 0.01 was tested well. Measured in meters.
double satSimDelta() { 
  return 0.01;
}

// Assemble the cam2world matrix from the along track, across track, and down vectors
// Note how we swap the first two columns and flip one sign. We went the along
// direction to be the camera y direction
void assembleCam2WorldMatrix(vw::Vector3 const& along, 
                             vw::Vector3 const& across, 
                             vw::Vector3 const& down,
                             // Output
                             vw::Matrix3x3 & cam2world) {

  for (int row = 0; row < 3; row++) {
    cam2world(row, 0) = along[row];
    cam2world(row, 1) = across[row];
    cam2world(row, 2) = down[row];
  }
 return;
}

// Return the matrix of rotation in the xy plane, from camera to satellite body
vw::Matrix3x3 rotationXY() {

  vw::Matrix3x3 T;
  // Set all elements to zero
  for (int row = 0; row < 3; row++)
    for (int col = 0; col < 3; col++)
      T(row, col) = 0.0;
  
  T(0, 1) = 1;
  T(1, 0) = -1;
  T(2, 2) = 1;

  return T;
}

// Given two end points in projected coordinates determining a satellite 
// trajectory, find the along and across vectors in projected coordinates.
void calcProjAlongAcross(vw::Vector3 const& first_proj,
                         vw::Vector3 const& last_proj,
                         vw::Vector3      & proj_along,
                         vw::Vector3      & proj_across) {

  proj_along = last_proj - first_proj;
  
  // Sanity check. 
  if (proj_along == vw::Vector3())
    vw::vw_throw(vw::ArgumentErr()
       << "The first and last camera positions are the same. It is not possible find the camera orientation. Specify at the very least two positions that are close but distinct.\n");

  // Normalize
  proj_along = proj_along / norm_2(proj_along);

  // One more sanity check
  if (std::max(std::abs(proj_along[0]), std::abs(proj_along[1])) < 1e-6)
    vw::vw_throw(vw::ArgumentErr()
      << "It appears that the satellite is aiming for the ground or "
      << "the orbital segment is too short. Correct the orbit end points.\n");

  // Find the across-track direction, parallel to the ground, in projected coords
  proj_across = vw::math::cross_prod(proj_along, vw::Vector3(0, 0, 1));
  proj_across = proj_across / norm_2(proj_across);
}

// Make these vectors have norm 1, and make across perpendicular to along
void normalizeOrthogonalizeAlongAcross(vw::Vector3 & along, vw::Vector3 & across) {
    
  // Normalize
  along = along / norm_2(along);
  across = across / norm_2(across);

  // Ensure that across is perpendicular to along
  across = across - dot_prod(along, across) * along;

  // Normalize again
  across = across / norm_2(across);
}

// Find normalized along and across directions in ECEF given these values in
// projected coordinates. Use centered difference to compute the along and
// across track points This achieves higher quality results.
void calcEcefAlongAcross(vw::cartography::GeoReference const& dem_georef,
                         double delta,
                         vw::Vector3 const& proj_along,
                         vw::Vector3 const& proj_across,
                         vw::Vector3 & proj_pt,
                         // Outputs
                         vw::Vector3 & along,
                         vw::Vector3 & across) {

  // Check if proj_along and proj_across are perpendicular and have norm 1
  double dot = dot_prod(proj_along, proj_across);
  if (std::abs(dot) > 1e-6) 
    vw::vw_throw(vw::ArgumentErr()
      << "calcEcefAlongAcross: proj_along and proj_across are not perpendicular.\n");
  if (std::abs(norm_2(proj_along)  - 1.0) > 1e-6 || 
      std::abs(norm_2(proj_across) - 1.0) > 1e-6) 
    vw::vw_throw(vw::ArgumentErr()
      << "calcEcefAlongAcross: either proj_along or proj_across does not have norm 1.\n");

  vw::Vector3 L1 = proj_pt - delta * proj_along; // along track point
  vw::Vector3 C1 = proj_pt - delta * proj_across; // across track point
  vw::Vector3 L2 = proj_pt + delta * proj_along; // along track point
  vw::Vector3 C2 = proj_pt + delta * proj_across; // across track point

  // Convert to cartesian
  L1 = vw::cartography::projToEcef(dem_georef, L1);
  C1 = vw::cartography::projToEcef(dem_georef, C1);
  L2 = vw::cartography::projToEcef(dem_georef, L2);
  C2 = vw::cartography::projToEcef(dem_georef, C2);

  // Create the along track and across track vectors
  along  = L2 - L1;
  across = C2 - C1;

  // Make these vector have norm 1, and make across perpendicular to along
  asp::normalizeOrthogonalizeAlongAcross(along, across);
}

} // end namespace asp
