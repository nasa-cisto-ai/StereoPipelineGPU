#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


// ===========================================================================
// The polygon rasterization code is based on s_pgdraw.c and
// s_pgspan.c from the OpenGL reference implementation
// ===========================================================================


// ===========================================================================
// NOTE: although within a pixel, we currently don't get results
// identical to NVIDIA's OpenGL implementation OpenGL version. This
// may be a difference between the definitions of the pixel center
// (NVIDIA may use the 0.5 pixel offset). We also may be using a
// subtly different transform to the final window coords. Should check
// againt __glUpdateViewport and __glUpdateViewportTransform in
// s_xform.c. Also code in __glim_Ortho. -- LJE
// ===========================================================================

// ===========================================================================
// According to Levoy:
//
//     http://graphics.stanford.edu/courses/cs248-01/scan/scan1.html
//
// SGI "uses a DDA with fixed-point arithmetic. log2N bits of
// sub-pixel precision for an N-pixel line". This seems to be similar
// to the approach taken here for polygon rasterization.
// ===========================================================================

#include <vw/Core/Exception.h>
#include <vw/Core/FundamentalTypes.h>
#include <asp/Core/SoftwareRenderer.h>

#include <iostream>

using namespace std;
using namespace vw;
using namespace stereo;

enum { eShadeFlat = 0x0, eShadeSmooth = 0x1 };

// ===========================================================================
// Constants and enums
// ===========================================================================

// NOTE: double does *NOT* work!
// #define _VW_USE_DOUBLE_
#ifdef _VW_USE_DOUBLE_

#define __TWO_63 9223372036854775808.0
#define __FRACTION(result, f) result = (vw::int64) ((f) * __TWO_63)
#define SHIFT 63
typedef double RealT;

#else

#define __TWO_31 2147483648.0F
#define __FRACTION(result, f) result = (vw::int32) ((f) * __TWO_31)
#define SHIFT 31
typedef float RealT;

#endif

static const int kVerticesPerTriangle = 3;

// ===========================================================================
// Type declarations
// ===========================================================================

// Coordinate structure.  Coordinates contain x, y, z and w.
struct Coords
{
  Coords() {}
  Coords(float coords[2])
  {
    x = coords[0]; y = coords[1]; z = 0.0; w = 1.0;
  }
  RealT x, y, z, w;
};

// Color structure.  Colors are composed of red, green, blue and alpha.
struct Color
{
  Color() {}
  Color(float gray) { r = gray; g = 0.0; b = 0.0; a = 1.0; }
  Color(float color[], int numComponents)
  {
    switch (numComponents)
    {
    case 1:
      r = color[0]; g = 0.0; b = 0.0; a = 1.0;
      return;
    case 3:
      r = color[0]; g = color[1]; b = color[2]; a = 1.0;
      return;
    case 4:
      r = color[0]; g = color[1]; b = color[2]; a = color[3];
      return;
    default:
      r = 0.0; g = 0.0; b = 0.0; a = 1.0;
    }
  }
  RealT r, g, b, a;
};

// Interpolator record for color interpolators.  Used for primary and
// secondary colors;
struct ColorIterator
{
  RealT rLittle, gLittle, bLittle, aLittle;
  RealT rBig, gBig, bBig, aBig;
  RealT drdx, dgdx, dbdx, dadx;
  RealT drdy, dgdy, dbdy, dady;
};

// A fragment is a collection of all the data needed after
// rasterization of a primitive has occurred, but before the data is
// entered into various framebuffers.  The data contained in the
// fragment has been normalized into a form for immediate storage into
// the framebuffer.

struct FragmentInfo
{
  FragmentInfo() {}
  int x, y;                                // Screen x, y
  // Colors of the fragment.  When in colorIndexMode only the r
  // component is valid.
  Color color;
};

// Shader record for iterated objects (lines/triangles).  This keeps
// track of all the various deltas needed to rasterize a triangle.
struct RasterInfo
{
  RasterInfo() {}
  int dxLeftLittle, dxLeftBig;
  int dxLeftFrac;
  int ixLeft, ixLeftFrac;

  int dxRightLittle, dxRightBig;
  int dxRightFrac;
  int ixRight, ixRightFrac;

  RealT area;
  RealT dxAC, dxBC, dyAC, dyBC;

  bool ccw;

  FragmentInfo frag;
  int length;

  ColorIterator  colorIter;

  unsigned int modeFlags;
};

struct GraphicsState
{
  GraphicsState() {}
  float *buffer;
  int numPixels;
  int width;
  int height;

  RasterInfo rasterInfo;
  Color currentFlatColor;
  // The smallest rectangle that is the intersection of the window clip
  // and the scissor clip.  If the scissor box is disabled then this
  // is just the window box. Note that the x0,y0 point is inside the
  // box but that the x1,y1 point is just outside the box.
  int clipX0, clipY0, clipX1, clipY1;
  // The viewport translated into offset window coordinates.  maxx and
  // maxy are one past the edge (an x coord is in if minx <= x <
  // maxx).
  int minx, miny, maxx, maxy;
};

struct Vertex
{
  Vertex(float initCoords[2], float initGray)
    : color(initGray), window(initCoords) {}
  Vertex(float initCoords[2], Color initColor)
    : color(initColor), window(initCoords) {}

  // Current face color in use.
  Color color;
  // Window coordinate. This field is filled in when the window clip
  // coordinate is converted to a drawing surface relative "window"
  // coordinate. NOTE: the window.w coordinate contains 1/clip.w.
  Coords window;
};


// ===========================================================================
// Functions
// ===========================================================================

inline RealT
TriangleArea(RealT dxAC, RealT dxBC, RealT dyAC, RealT dyBC)
{
  RealT area = dxAC * dyBC - dxBC * dyAC;
  return area;
}

static void
SortVertices(Vertex * &a, Vertex * &b, Vertex * &c)
{
  if (*(int *)&a->window.y < *(int *) &b->window.y)
  {
    if (*(int *)&b->window.y < *(int *) &c->window.y)      // Already sorted
    {
      return;
    }
    else
    {
      if (*(int *)&a->window.y < *(int *)&c->window.y)
      {
        Vertex *temp = b;
        b = c;
        c = temp;
      }
      else
      {
        Vertex *temp = a;
        a = c;
        c = b;
        b = temp;
      }
    }
  }
  else
  {
    if (*(int *)&b->window.y < *(int *)&c->window.y)
    {
      if (*(int *)&a->window.y < *(int *)&c->window.y)
      {
        Vertex *temp = a;
        a = b;
        b = temp;
      }
      else
      {
        Vertex *temp = a;
        a = b;
        b = c;
        c = temp;
      }
    }
    else
    {
      Vertex *temp = a;
      a = c;
      c = temp;
    }
   }
}

static void
SetInitialParameters(GraphicsState *gc, const Vertex *a, RealT dx, RealT dy)
{
  RasterInfo *rasterInfo = &gc->rasterInfo;
  RealT big = rasterInfo->dxLeftBig;
  RealT little = rasterInfo->dxLeftLittle;
  const Color *vertexColor;
  Color *fragColor;
  ColorIterator *iter;
  unsigned int modeFlags = rasterInfo->modeFlags;

  if (big > little)
  {
    if (modeFlags & eShadeSmooth)
    {
      vertexColor = &a->color;
      fragColor = &rasterInfo->frag.color;
      iter = &rasterInfo->colorIter;

      fragColor->r = vertexColor->r + dx*iter->drdx + dy*iter->drdy;
      iter->rLittle = iter->drdy + little * iter->drdx;
      iter->rBig = iter->rLittle + iter->drdx;
    }
  }
  else
  {
    if (modeFlags & eShadeSmooth)
    {
      vertexColor = &a->color;
      fragColor = &rasterInfo->frag.color;
      iter = &rasterInfo->colorIter;

      fragColor->r = vertexColor->r + dx*iter->drdx + dy*iter->drdy;
      iter->rLittle = iter->drdy + little * iter->drdx;
      iter->rBig = iter->rLittle - iter->drdx;
    }
  }
}


static void
DrawFlatGraySpan(GraphicsState *gc)
{
  // Evaluate the clipping in the X direction
  int length = gc->rasterInfo.length;
  int x = gc->rasterInfo.frag.x;
  if ( x < gc->clipX0 ) { // Check to see if the line goes off the left
    length += x - gc->clipX0;
    x = gc->clipX0;
  }
  if ( x + length > gc->clipX1 ) { // Check to see the line goes off
                                   // the right
    length -= x + length - gc->clipX1;
  }

  // Check to see if we just removed this line
  if ( length < 1 ) return;

  std::fill_n(&(gc->buffer[gc->rasterInfo.frag.y * gc->width + x]),
              length, float(gc->rasterInfo.frag.color.r));
}

static void
DrawGraySpan(GraphicsState *gc) {
  RealT gray = gc->rasterInfo.frag.color.r;
  RealT drdx = gc->rasterInfo.colorIter.drdx;

  // Evaluate the clipping in the X direction
  int length = gc->rasterInfo.length;
  int x = gc->rasterInfo.frag.x;
  if ( x < gc->clipX0 ) { // Check to see if the line goes off the left
    int difference = gc->clipX0 - x;
    length -= difference;
    x = gc->clipX0;
    gray += RealT(difference) * drdx;
  }
  if ( x + length > gc->clipX1 ) { // Check to see the line goes off
                                   // the right
    length -= x + length - gc->clipX1;
  }

  // Check to see if we just removed this line
  if ( length < 1 ) return;

  float *span = &(gc->buffer[gc->rasterInfo.frag.y * gc->width + x ] );

  for (int i = length; i; --i) {
    *span++ = float(gray);
    gray += drdx;
  }
}

// In the SnapX* and FillSubTriangle routines, 1s31.1s31 fixed point
// arithmetic is used with the integer and fractional portions carried
// in separate ints (e.g., ixLeft and ixLeftFrac)
static void
SnapXLeft(GraphicsState *gc, RealT xLeft, RealT dxdyLeft)
{
  RealT little, dx;
  int ixLeft, ixLeftFrac, frac, ilittle, ibig;

  ixLeft = (int) xLeft;
  dx = xLeft - ixLeft;
  __FRACTION(ixLeftFrac, dx);

  // Is the following right? Seems like it should be ixLeft - (...) --LJE
  gc->rasterInfo.ixLeft = ixLeft + (((unsigned int) ixLeftFrac) >> SHIFT);
  gc->rasterInfo.ixLeftFrac = ixLeftFrac & ~0x80000000;

  // Compute big and little steps
  ilittle = (int) dxdyLeft;
  little = (RealT) ilittle;
  if (dxdyLeft < 0) {
    ibig = ilittle - 1;
    dx = little - dxdyLeft;
    // Here we know ilittle and ibig are not going to be zero, and
    // that frac will be negative, so we can just negate frac, to get
    // things in our standard form --LJE
    __FRACTION(frac, dx);
    gc->rasterInfo.dxLeftFrac = -frac;
  } else {
    ibig = ilittle + 1;
    dx = dxdyLeft - little;
    __FRACTION(frac, dx);
    gc->rasterInfo.dxLeftFrac = frac;
  }

  gc->rasterInfo.dxLeftLittle = ilittle;
  gc->rasterInfo.dxLeftBig = ibig;
}

static void
SnapXRight(RasterInfo *rasterInfo, RealT xRight, RealT dxdyRight)
{
  RealT little, big, dx;
  int ixRight, ixRightFrac, frac;

  ixRight = (int) xRight;
  dx = xRight - ixRight;
  __FRACTION(ixRightFrac, dx);

  // Is the following right? Seems like it should be ixRight - (...) -LJE
  rasterInfo->ixRight = ixRight + (((unsigned int) ixRightFrac) >> SHIFT);
  rasterInfo->ixRightFrac = ixRightFrac & ~0x80000000;

  // Compute big and little steps
  little = (RealT) ((int) dxdyRight);

  if (dxdyRight < 0)
  {
    big = little - 1;
    dx = little - dxdyRight;
    __FRACTION(frac,dx);
    rasterInfo->dxRightFrac = -frac;
  }
  else
  {
    big = little + 1;
    dx = dxdyRight - little;
    __FRACTION(frac,dx);
    rasterInfo->dxRightFrac = frac;
  }

  rasterInfo->dxRightLittle = (int) little;
  rasterInfo->dxRightBig = (int) big;
}

static void
FillSubTriangle(GraphicsState *gc,  int iyBottom, int iyTop)
{
  int ixLeft, ixRight;
  int ixLeftFrac, ixRightFrac;
  int dxLeftFrac, dxRightFrac;
  int dxLeftLittle, dxRightLittle;
  int dxLeftBig, dxRightBig;
  int spanWidth, clipY0, clipY1;
  unsigned int modeFlags;
  void (*processSpan)(GraphicsState *);

  ixLeft = gc->rasterInfo.ixLeft;
  ixLeftFrac = gc->rasterInfo.ixLeftFrac;
  ixRight = gc->rasterInfo.ixRight;
  ixRightFrac = gc->rasterInfo.ixRightFrac;
  clipY0 = gc->clipY0;
  clipY1 = std::min( gc->clipY1, iyTop );
  dxLeftFrac = gc->rasterInfo.dxLeftFrac;
  dxLeftBig = gc->rasterInfo.dxLeftBig;
  dxLeftLittle = gc->rasterInfo.dxLeftLittle;
  dxRightFrac = gc->rasterInfo.dxRightFrac;
  dxRightBig = gc->rasterInfo.dxRightBig;
  dxRightLittle = gc->rasterInfo.dxRightLittle;
  modeFlags = gc->rasterInfo.modeFlags;
  //  gc->rasterInfo.cfb = gc->drawBuffer;

  if (modeFlags & eShadeSmooth)
    processSpan = DrawGraySpan;
  else
    processSpan = DrawFlatGraySpan;

  while (iyBottom < clipY1)
  {
    spanWidth = ixRight - ixLeft;
    // Only render spans that have non-zero width and which are not
    // scissored out vertically.
    if ((spanWidth > 0) && (iyBottom >= clipY0) ) {
      gc->rasterInfo.frag.x = ixLeft;
      gc->rasterInfo.frag.y = iyBottom;
      gc->rasterInfo.length = spanWidth;
      processSpan(gc);
    }

    // Advance right edge fixed point, adjusting for carry
    ixRightFrac += dxRightFrac;
    if (ixRightFrac < 0)            // Carry/Borrow'd. Use large step
    {
      ixRight += dxRightBig;
      ixRightFrac &= ~0x80000000;
    }
    else                            // Use small step
    {
      ixRight += dxRightLittle;
    }

    iyBottom++;
    ixLeftFrac += dxLeftFrac;
    if (ixLeftFrac < 0)             // Carry/Borrow'd. Use large step
    {
      ixLeft += dxLeftBig;
      ixLeftFrac &= ~0x80000000;

      if (modeFlags & eShadeSmooth)
        gc->rasterInfo.frag.color.r += gc->rasterInfo.colorIter.rBig;
    }
    else                            // Use small step
    {
      ixLeft += dxLeftLittle;
      if (modeFlags & eShadeSmooth)
        gc->rasterInfo.frag.color.r += gc->rasterInfo.colorIter.rLittle;
    }
  }
  gc->rasterInfo.ixLeft = ixLeft;
  gc->rasterInfo.ixLeftFrac = ixLeftFrac;
  gc->rasterInfo.ixRight = ixRight;
  gc->rasterInfo.ixRightFrac = ixRightFrac;
}

static void FillTriangle(GraphicsState *gc, Vertex *a, Vertex *b, Vertex *c) {
  RealT area, oneOverArea, t1, t2, t3, t4;
  RealT dxAC, dxBC, dyAC, dyBC;
  RealT dxAB, dyAB;
  RealT dx, dy, dxdyLeft, dxdyRight;
  Color *aColor, *bColor;
  int aIY, bIY, cIY;
  unsigned int modeFlags;
  bool ccw;                             // was a float for some reason

  // Sort vertices in y.
  SortVertices(a, b, c);

  // Set edge vectors
  dxAC = a->window.x - c->window.x;
  dxBC = b->window.x - c->window.x;
  dyAC = a->window.y - c->window.y;
  dyBC = b->window.y - c->window.y;

  // Compute signed area of the triangle
  area = TriangleArea(dxAC, dxBC, dyAC, dyBC);
  ccw = (area >= 0);   //  ccw = !(* (int *) & area >> 31);

  // Pre-compute one over polygon area
  oneOverArea = (area != 0) ? (1.0 / area) : 0.0;

  // Fetch some stuff we are going to reuse
  modeFlags = gc->rasterInfo.modeFlags;
  aColor = &a->color;                      // & -LJE
  bColor = &b->color;                      // & -LJE

  // Compute delta values for unit changes in x or y for each
  // parameter.
  t1 = dyAC * oneOverArea;
  t2 = dyBC * oneOverArea;
  t3 = dxAC * oneOverArea;
  t4 = dxBC * oneOverArea;

  if (modeFlags & eShadeSmooth)
  {
    RealT drAC, drBC;
    Color *cColor = &c->color;             // & -LJE

    // If gray scale the intensity is carried in the red component
    drAC = aColor->r - cColor->r;
    drBC = bColor->r - cColor->r;
    gc->rasterInfo.colorIter.drdx = drAC * t2 - drBC * t1;
    gc->rasterInfo.colorIter.drdy = drBC * t3 - drAC * t4;
  }
  else
  {
    Color *flatColor = &gc->currentFlatColor; // & -LJE
    gc->rasterInfo.frag.color.r = flatColor->r;
  }

  // Snap each y coordinate to its pixel center
  aIY = (int) (a->window.y);
  bIY = (int) (b->window.y);
  cIY = (int) (c->window.y);

  // This algorithim always fills from bottom to top, left to right.
  // Because of this, ccw triangles are inherently faster because the
  // parameter values need not be recomputed.
  dxAB = a->window.x - b->window.x;
  dyAB = a->window.y - b->window.y;

  if (ccw)
  {
    dxdyLeft = (dyAC != 0.0) ? dxAC / dyAC : 0.0;
    dy = (aIY + 1.0) - a->window.y;
    SnapXLeft(gc, a->window.x + dy*dxdyLeft, dxdyLeft);
    dx = (gc->rasterInfo.ixLeft + 1.0) - a->window.x;
    SetInitialParameters(gc, a, dx, dy);
    if (aIY != bIY)
    {
      dxdyRight = (dyAB != 0.0) ? dxAB / dyAB : 0.0;
      SnapXRight(&gc->rasterInfo, a->window.x + dy*dxdyRight, dxdyRight);
      FillSubTriangle(gc, aIY, bIY);
    }
    if (bIY != cIY)
    {
      dxdyRight = (dyBC != 0.0) ? dxBC / dyBC : 0.0;
      dy = (bIY + 1.0) - b->window.y;
      SnapXRight(&gc->rasterInfo, b->window.x + dy*dxdyRight, dxdyRight);
      FillSubTriangle(gc, bIY, cIY);
    }
  }
  else
  {
    dxdyRight = (dyAC != 0.0) ? dxAC / dyAC : 0.0;
    dy = (aIY + 1.0) - a->window.y;
    SnapXRight(&gc->rasterInfo, a->window.x + dy*dxdyRight, dxdyRight);
    if (aIY != bIY)
    {
      dxdyLeft = (dyAB != 0.0) ? dxAB / dyAB : 0.0;
      SnapXLeft(gc, a->window.x + dy*dxdyLeft, dxdyLeft);
      dx = (gc->rasterInfo.ixLeft + 1.0) - a->window.x;
      SetInitialParameters(gc, a, dx, dy);
      FillSubTriangle(gc, aIY, bIY);
    }
    if (bIY != cIY)
    {
      dxdyLeft = (dyBC != 0.0) ? dxBC / dyBC : 0.0;
      dy = (bIY + 1.0) - b->window.y;
      SnapXLeft(gc, b->window.x + dy*dxdyLeft, dxdyLeft);
      dx = (gc->rasterInfo.ixLeft + 1.0) - b->window.x;
      SetInitialParameters(gc, b, dx, dy);
      FillSubTriangle(gc, bIY, cIY);
    }
  }
}

inline void
MapToWindow(Coords &coords,
            const double ndcMap[3][2],
            double /*x0*/, double /*y0*/, double width, double height,
            Coords &result)
{
  // The separation of NDC and viewport mappings might seem silly
  // right now, but it allows one to modify the projection matrix
  // independently of the final map to window coords and vice
  // versa. Also if we ever do clipping it will be useful.

  // According to the glViewPort specification:
  // xw = (xNDC + 1)(width/2) + x
  // yw = (xNDC + 1)(height/2) + y
  double xNDC = ndcMap[0][0] * coords.x + ndcMap[2][0];
  double yNDC = ndcMap[1][1] * coords.y + ndcMap[2][1];

  result.x = 0.5 * (xNDC + 1.0) * width;
  result.y = 0.5 * (yNDC + 1.0) * height;
}


// ===========================================================================
// Class Member Functions
// ===========================================================================

SoftwareRenderer::SoftwareRenderer(const int width, const int height, float *buffer)
{
  m_numVertexComponents = 0;
  m_vertexPointer = 0;
  m_triangleVertexStep = 0;

  m_numColorComponents = 0;
  m_colorPointer = 0;
  m_triangleColorStep = 0;

  m_bufferWidth = width;
  m_bufferHeight = height;
  m_buffer = buffer;

  double deltaX = double(m_bufferWidth);
  double deltaY = double(m_bufferHeight);

  m_transformViewport[0][1] = m_transformViewport[1][0] = 0.0;
  // scaling
  m_transformViewport[0][0] = deltaX * 0.5;
  m_transformViewport[1][1] = deltaY * 0.5;
  // translation
  m_transformViewport[2][0] = deltaX * 0.5;
  m_transformViewport[2][1] = deltaY * 0.5;

  m_transformNDC[0][1] = m_transformNDC[1][0] = 0.0;
  // scaling
  m_transformNDC[0][0] = 2.0 / deltaX;
  m_transformNDC[1][1] = 2.0 / deltaY;
  // translation
  m_transformNDC[2][0] = -1.0;
  m_transformNDC[2][1] = -1.0;

  m_shadeMode = eShadeSmooth;
  m_currentFlatColor[0] = m_currentFlatColor[1] = m_currentFlatColor[2] = 0.0;

  // FIX ME!!! Put this state info in the software renderer class
  GraphicsState *graphicsState = new GraphicsState;
  graphicsState->buffer = m_buffer;
  graphicsState->currentFlatColor.r = m_currentFlatColor[0];
  graphicsState->width = m_bufferWidth;
  graphicsState->height = m_bufferHeight;
  graphicsState->rasterInfo.modeFlags = eShadeSmooth;
  graphicsState->minx = graphicsState->miny = 0;
  graphicsState->maxx = m_bufferWidth;
  graphicsState->maxy = m_bufferHeight;
  graphicsState->clipX0 = graphicsState->clipY0 = 0;
  graphicsState->clipX1 = m_bufferWidth;
  graphicsState->clipY1 = m_bufferHeight;
  m_graphicsState = graphicsState;
}

// Free up resources that are allocated in the constructor
SoftwareRenderer::~SoftwareRenderer() {

  if (m_graphicsState)
    delete static_cast<GraphicsState*>(m_graphicsState);

}

void
SoftwareRenderer::Ortho2D(const double left, const double right,
                          const double bottom, const double top)
{
  // This takes a 2D point to normalized coordinates (i.e, -1.0 <= x <= 1.0)
  //
  //   [2/(r-l)    0    -(r+l)/(r-l)]
  //   [   0    2/(t-b) -(t+b)/(t-b)]
  //   [   0       0         1     ]
  double deltax = right - left;
  double deltay = top - bottom;

  if ((deltax == 0.0) || (deltay == 0.0))
    vw_throw(LogicErr() << "SoftwareRenderer: Ortho2D failed.  Projection dimensions are zero.");

  m_transformNDC[0][1] = m_transformNDC[1][0] = 0.0;
  m_transformNDC[0][0] = 2.0 / deltax;
  m_transformNDC[1][1] = 2.0 / deltay;
  m_transformNDC[2][0] = -(right + left) / deltax;
  m_transformNDC[2][1] = -(top + bottom) / deltay;
}

void
SoftwareRenderer::Clear(const float value) {
  int bufferSize = m_bufferWidth * m_bufferHeight;
  for (int i = 0; i < bufferSize; ++i)
    m_buffer[i] = value;
}

void
SoftwareRenderer::SetVertexPointer(const int numComponents, float * const vertices)
{
  m_numVertexComponents = numComponents;
  m_triangleVertexStep = kVerticesPerTriangle * m_numVertexComponents;

  m_vertexPointer = vertices;
}

void
SoftwareRenderer::SetColorPointer(const int numComponents, float * const colors)
{
  m_numColorComponents = numComponents;
  m_triangleColorStep = kVerticesPerTriangle * m_numColorComponents;

  m_colorPointer = colors;
}

void
SoftwareRenderer::DrawPolygon(const int startIndex, const int numVertices) {
  if (m_vertexPointer == 0)
    return;

  if ((m_colorPointer == 0) && (m_shadeMode != eShadeFlat))
    return;

  // NOTE: we assume polygons are convex! This allows one to easily
  // fan triangulate them.
  int numTriangles = numVertices - 2;
  float *vertices = &m_vertexPointer[startIndex * m_numVertexComponents];
  float *colors = &m_colorPointer[startIndex * m_numColorComponents];
  int vertexIndex1 = m_numVertexComponents;
  int vertexIndex2 = vertexIndex1 + m_numVertexComponents;
  int colorIndex1 = m_numColorComponents;
  int colorIndex2 = colorIndex1 + m_numColorComponents;
  ::Color color0(&colors[0], m_numColorComponents);
  Vertex vertex0(vertices, color0);

  MapToWindow(vertex0.window, m_transformNDC,
              0.0, 0.0, double(m_bufferWidth), double(m_bufferHeight),
              vertex0.window);

  for (int i = 0; i < numTriangles; i++)
  {
    ::Color color1(&colors[colorIndex1], m_numColorComponents);
    ::Color color2(&colors[colorIndex2], m_numColorComponents);

    Vertex vertex1(&vertices[vertexIndex1], color1);
    Vertex vertex2(&vertices[vertexIndex2], color2);

    MapToWindow(vertex1.window, m_transformNDC,
                0.0, 0.0, double(m_bufferWidth), double(m_bufferHeight),
                vertex1.window);
    MapToWindow(vertex2.window, m_transformNDC,
                0.0, 0.0, double(m_bufferWidth), double(m_bufferHeight),
                vertex2.window);

    FillTriangle((GraphicsState *) m_graphicsState, &vertex0, &vertex1, &vertex2);

    vertexIndex1 += m_triangleVertexStep;
    vertexIndex2 += m_triangleVertexStep;
    colorIndex1 += m_triangleColorStep;
    colorIndex2 += m_triangleColorStep;
  }
}
