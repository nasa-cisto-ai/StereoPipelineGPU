// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Math/LevenbergMarquardt.h>
#include <vw/Math/Matrix.h>
#include <vw/Camera/CameraModel.h>
#include <asp/IsisIO/IsisInterfaceLineScan.h>

// Isis headers
#include <Camera.h>
#include <CameraDetectorMap.h>
#include <CameraDistortionMap.h>
#include <CameraFocalPlaneMap.h>
#include <iTime.h>
#include <SurfacePoint.h>
#include <Latitude.h>
#include <Longitude.h>
#include <Angle.h>

#include <algorithm>
#include <vector>
#include <boost/smart_ptr/scoped_ptr.hpp>

using namespace vw;
using namespace asp;
using namespace asp::isis;

// Construct
IsisInterfaceLineScan::IsisInterfaceLineScan(std::string const& filename): IsisInterface(filename), m_alphacube(*m_cube) {

  // Gutting Isis::Camera
  m_distortmap = m_camera->DistortionMap();
  m_focalmap   = m_camera->FocalPlaneMap();
  m_detectmap  = m_camera->DetectorMap();
}

// Custom function to help avoid over invoking the deeply buried
// functions of Isis::Sensor
void IsisInterfaceLineScan::SetTime(Vector2 const& px, bool calc_pose) const {
  if (px != m_c_location) {
    m_c_location = px;
    m_detectmap->SetParent(m_alphacube.AlphaSample(px[0]),
                           m_alphacube.AlphaLine(px[1]));

    if (calc_pose) {
      // Calculating the spacecraft position and orientation (hence pose)
      m_camera->instrumentPosition(&m_center[0]);
      m_center *= 1000;

      std::vector<double> rot_inst = m_camera->instrumentRotation()->Matrix();
      std::vector<double> rot_body = m_camera->bodyRotation()->Matrix();
      MatrixProxy<double,3,3> R_inst(&(rot_inst[0]));
      MatrixProxy<double,3,3> R_body(&(rot_body[0]));
      m_pose = Quat(R_body*transpose(R_inst));
    }
  }
}

class EphemerisLMA : public vw::math::LeastSquaresModelBase<EphemerisLMA> {
  vw::Vector3 m_point;
  Isis::Camera* m_camera;
  Isis::CameraDistortionMap *m_distortmap;
  Isis::CameraFocalPlaneMap *m_focalmap;
public:
  typedef vw::Vector<double> result_type; // Back project result
  typedef vw::Vector<double> domain_type; // Ephemeris time
  typedef vw::Matrix<double> jacobian_type;

  inline EphemerisLMA(vw::Vector3 const& point,
                      Isis::Camera* camera,
                      Isis::CameraDistortionMap* distortmap,
                      Isis::CameraFocalPlaneMap* focalmap) : m_point(point), m_camera(camera), m_distortmap(distortmap), m_focalmap(focalmap) {}

  inline result_type operator()(domain_type const& x) const;
};


// LMA for projecting point to linescan camera
EphemerisLMA::result_type
EphemerisLMA::operator()(EphemerisLMA::domain_type const& x) const {

  // Setting Ephemeris Time
  m_camera->setTime(Isis::iTime(x[0]));

  // Calculating the look direction in camera frame
  Vector3 instru;
  m_camera->instrumentPosition(&instru[0]);
  instru *= 1000;  // Spice gives in km
  Vector3 lookB = normalize(m_point - instru);
  std::vector<double> lookB_copy(3);
  std::copy(lookB.begin(), lookB.end(), lookB_copy.begin());
  std::vector<double> lookJ = m_camera->bodyRotation()->J2000Vector(lookB_copy);
  std::vector<double> lookC = m_camera->instrumentRotation()->ReferenceVector(lookJ);
  Vector3 look;
  std::copy(lookC.begin(), lookC.end(), look.begin());

  // Projecting to mm focal plane
  look = m_camera->FocalLength() * (look / look[2]);
  m_distortmap->SetUndistortedFocalPlane(look[0], look[1]);
  m_focalmap->SetFocalPlane(m_distortmap->FocalPlaneX(),
                            m_distortmap->FocalPlaneY());
  result_type result(1);
  // Not exactly sure about lineoffset .. but ISIS does it
  result[0] = m_focalmap->DetectorLineOffset() - m_focalmap->DetectorLine();

  return result;
}

Vector2
IsisInterfaceLineScan::point_to_pixel(Vector3 const& point) const {

#if 1
   // First seed LMA with an ephemeris time in the middle of the image
  double middle = lines() / 2;
  m_detectmap->SetParent(1, m_alphacube.AlphaLine(middle));
  double start_e = m_camera->time().Et();

  // Build LMA
  EphemerisLMA model(point, m_camera.get(), m_distortmap, m_focalmap);
  int status;
  Vector<double> objective(1), start(1);
  start[0] = start_e;
  Vector<double> solution_e = math::levenberg_marquardt(model,
                                                        start,
                                                        objective,
                                                        status);

  // Make sure we found ideal time
  VW_ASSERT(status > 0, vw::camera::PointToPixelErr()
            << " Unable to project point into ISIS linescan camera ");

  // Converting now to pixel
  m_camera->setTime(Isis::iTime(solution_e[0]));

  // Working out pointing
  m_camera->instrumentPosition(&m_center[0]);
  m_center *= 1000;
  Vector3 look = normalize(point-m_center);

  // Calculating rotation to camera frame
  std::vector<double> rot_inst = m_camera->instrumentRotation()->Matrix();
  std::vector<double> rot_body = m_camera->bodyRotation()->Matrix();
  MatrixProxy<double,3,3> R_inst(&(rot_inst[0]));
  MatrixProxy<double,3,3> R_body(&(rot_body[0]));
  m_pose = Quat(R_body*transpose(R_inst));

  look = inverse(m_pose).rotate(look);
  look = m_camera->FocalLength() * (look / look[2]);
  m_distortmap->SetUndistortedFocalPlane(look[0], look[1]);
  m_focalmap->SetFocalPlane(m_distortmap->FocalPlaneX(),
                            m_distortmap->FocalPlaneY());
  m_detectmap->SetDetector(m_focalmap->DetectorSample(),
                           m_focalmap->DetectorLine());
  Vector2 pixel(m_detectmap->ParentSample(),
                m_detectmap->ParentLine());
  pixel[0] = m_alphacube.BetaSample(pixel[0]);
  pixel[1] = m_alphacube.BetaLine(pixel[1]);
  SetTime(pixel, false);

  pixel -= Vector2(1,1);
  return pixel;

#else
  // TODO(oalexan1): This code looks buggy. Need to set the height above ground too.
  Vector3 llh = m_datum.cartesian_to_geodetic(point);
  if (llh[0] < 0)
    llh[0] += 360.0;
  
  if (!m_camera->SetGround(Isis::Latitude (llh[1], Isis::Angle::Degrees),
                           Isis::Longitude(llh[0], Isis::Angle::Degrees)))
    vw_throw(camera::PixelToRayErr() << "Failed in SetGround().");
  
  return Vector2(m_camera->Sample() - 1.0, m_camera->Line() - 1.0);
#endif
  
}

Vector3
IsisInterfaceLineScan::pixel_to_vector(Vector2 const& pix) const {
  Vector2 px = pix + Vector2(1,1);
  SetTime(px, true);

  // Projecting to get look direction
  Vector3 result;
  m_focalmap->SetDetector(m_detectmap->DetectorSample(),
                          m_detectmap->DetectorLine());
  m_distortmap->SetFocalPlane(m_focalmap->FocalPlaneX(),
                              m_focalmap->FocalPlaneY());
  result[0] = m_distortmap->UndistortedFocalPlaneX();
  result[1] = m_distortmap->UndistortedFocalPlaneY();
  result[2] = m_distortmap->UndistortedFocalPlaneZ();
  result = normalize(result);
  result = m_pose.rotate(result);
  return result;
}

Vector3
IsisInterfaceLineScan::camera_center(Vector2 const& pix) const {
  Vector2 px = pix + Vector2(1,1);
  SetTime(px, true);
  return m_center;
}

Quat
IsisInterfaceLineScan::camera_pose(Vector2 const& pix) const {
  Vector2 px = pix + Vector2(1,1);
  SetTime(px, true);
  return m_pose;
}
