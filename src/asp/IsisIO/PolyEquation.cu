#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Core/FundamentalTypes.h>
#include <vw/Math/Vector.h>
#include <asp/IsisIO/PolyEquation.h>

#include <iomanip>
#include <vector>
#include <string>
#include <algorithm>

#include <boost/algorithm/string/split.hpp>
#include <boost/algorithm/string/classification.hpp>

using namespace vw;
using namespace asp;

// Constructors
//---------------------------------------------
PolyEquation::PolyEquation ( int order ) {
  if ( order < 0 )
    vw_throw( ArgumentErr() << "PolyEquation: Polynomial order must be greater than zero." );
  if ( order > 254 )
    vw_throw( ArgumentErr() << "PolyEquation: Polynomial order must be less than 255" );
  m_x_coeff.set_size( order + 1 );
  m_y_coeff.set_size( order + 1 );
  m_z_coeff.set_size( order + 1 );
  for ( int i = 0; i < order+1; i++ )
    m_x_coeff[i] = m_y_coeff[i] = m_z_coeff[i] = 0;
  m_cached_time = -1;
  m_time_offset = 0;
  m_max_length = uint8(order)+1;
}
PolyEquation::PolyEquation( int order_x,
                            int order_y,
                            int order_z ) {
  if ( order_x < 0 || order_y < 0 || order_z < 0 )
    vw_throw( ArgumentErr() << "PolyEquation: Polynomial order must be greater than zero." );
  if ( order_x > 254 || order_y > 254 || order_z > 254 )
    vw_throw( ArgumentErr() << "PolyEquation: Polynomial order must be less than 255" );
  m_x_coeff.set_size(order_x+1);
  m_y_coeff.set_size(order_y+1);
  m_z_coeff.set_size(order_z+1);
  for ( unsigned i = 0; i < m_x_coeff.size(); i++ )
    m_x_coeff[i] = 0;
  for ( unsigned i = 0; i < m_y_coeff.size(); i++ )
    m_y_coeff[i] = 0;
  for ( unsigned i = 0; i < m_z_coeff.size(); i++ )
    m_z_coeff[i] = 0;
  m_cached_time = -1;
  m_time_offset = 0;
  m_max_length = uint8( std::max( order_x, std::max( order_y, order_z ) ) ) + 1;
}

// Update
//-----------------------------------------------
void PolyEquation::update( double t ) {
  m_cached_time = t;
  double delta_t = t-m_time_offset;
  Vector<double> powers( m_max_length );
  powers[0] = 1;
  for ( uint8 i = 1; i < m_max_length; i++ )
    powers[i] = powers[i-1]*delta_t;
  m_cached_output[0] = sum( elem_prod(m_x_coeff,
                                      subvector(powers,0,m_x_coeff.size())) );
  m_cached_output[1] = sum( elem_prod(m_y_coeff,
                                      subvector(powers,0,m_y_coeff.size())) );
  m_cached_output[2] = sum( elem_prod(m_z_coeff,
                                      subvector(powers,0,m_z_coeff.size())) );
}

// FileIO
//-----------------------------------------------
void PolyEquation::write( std::ofstream& f ) {
  for ( int i = 0; i < 3; i++ ) {
    Vector<double>* pointer;
    switch(i) {
    case 0:
      pointer = &m_x_coeff;
      break;
    case 1:
      pointer = &m_y_coeff;
      break;
    default:
    case 2:
      pointer = &m_z_coeff;
      break;
    }

    f << std::setprecision( 15 );
    for ( unsigned j = 0; j < (*pointer).size(); j++ )
      f << (*pointer)[j] << " ";
    f << "\n";
  }
}

void PolyEquation::read( std::ifstream& f ) {
  std::string buffer;
  std::vector<std::string> tokens;
  m_cached_time = -1;
  for ( int i = 0; i < 3; i++ ) {
    buffer = "";
    std::getline( f, buffer );
    boost::split( tokens, buffer, boost::is_any_of(" =\n") );

    // Cleaning out any tokens that are just ""
    for(std::vector<std::string>::iterator iter = tokens.begin();
        iter != tokens.end(); ++iter )
      if ( (*iter) == "" ) {
        iter = tokens.erase(iter);
        iter--;
      }

    Vector<double>* pointer;
    switch(i) {
    case 0:
      pointer = &m_x_coeff;
      break;
    case 1:
      pointer = &m_y_coeff;
      break;
    default:
    case 2:
      pointer = &m_z_coeff;
      break;
    }

    pointer->set_size( tokens.size() );
    for ( unsigned j = 0; j < tokens.size(); j++ )
      (*pointer)[j] = atof( tokens[j].c_str() );

  }
}

// Constant Access
//-----------------------------------------------
double& PolyEquation::operator[]( size_t n ) {
  m_cached_time = -1;
  if (n >= m_x_coeff.size() + m_y_coeff.size() + m_z_coeff.size())
    vw_throw(ArgumentErr() << "PolyEquation: invalid index.");
  if (n < m_x_coeff.size()) {
    return m_x_coeff[n];
  } else if (n < m_x_coeff.size() + m_y_coeff.size()) {
    return m_y_coeff[n - m_x_coeff.size()];
  } else {
    return m_z_coeff[n - m_x_coeff.size() - m_y_coeff.size()];
  }
}
