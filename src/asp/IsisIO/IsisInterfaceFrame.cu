#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <vw/Math/Matrix.h>
#include <asp/IsisIO/IsisInterfaceFrame.h>

#include <algorithm>
#include <vector>

#include <Camera.h>
#include <CameraDetectorMap.h>
#include <CameraDistortionMap.h>
#include <CameraFocalPlaneMap.h>

using namespace vw;
using namespace asp;
using namespace asp::isis;

// Constructor
IsisInterfaceFrame::IsisInterfaceFrame(std::string const& filename):
  IsisInterface(filename), m_alphacube(*m_cube) {

  // Gutting Isis::Camera
  m_distortmap = m_camera->DistortionMap();
  m_focalmap   = m_camera->FocalPlaneMap();
  m_detectmap  = m_camera->DetectorMap();

  // Calculate the camera center (just once)
  m_camera->instrumentPosition(&m_center[0]);
  m_center *= 1000.0; // convert to meters

  // Calculating the camera pose (just once)
  std::vector<double> rot_inst = m_camera->instrumentRotation()->Matrix();
  std::vector<double> rot_body = m_camera->bodyRotation()->Matrix();
  MatrixProxy<double,3,3> R_inst(&(rot_inst[0]));
  MatrixProxy<double,3,3> R_body(&(rot_body[0]));

  // Instrument Rotation = Spacecraft to Camera's Frame
  // Body Rotation = Spacecraft to World Frame
  m_pose = Quat(R_body*transpose(R_inst));
}

Vector2
IsisInterfaceFrame::point_to_pixel(Vector3 const& point) const {

  Vector3 look = normalize(point - m_center);
  std::vector<double> lookB_copy(3);
  std::copy(look.begin(), look.end(), lookB_copy.begin());
  lookB_copy = m_camera->bodyRotation()->J2000Vector(lookB_copy);
  lookB_copy = m_camera->instrumentRotation()->ReferenceVector(lookB_copy);
  std::copy(lookB_copy.begin(), lookB_copy.end(), look.begin());
  look = m_camera->FocalLength() * (look / std::abs(look[2]));

  // Project into the camera
  m_distortmap->SetUndistortedFocalPlane(look[0], look[1]);
  m_focalmap->SetFocalPlane(m_distortmap->FocalPlaneX(),
                             m_distortmap->FocalPlaneY());
  m_detectmap->SetDetector(m_focalmap->DetectorSample(),
                            m_focalmap->DetectorLine());
  return Vector2(m_alphacube.BetaSample(m_detectmap->ParentSample()) - 1,
                  m_alphacube.BetaLine(m_detectmap->ParentLine()) - 1);
}

Vector3
IsisInterfaceFrame::pixel_to_vector(Vector2 const& px) const {
  m_detectmap->SetParent(m_alphacube.AlphaSample(px[0] + 1), m_alphacube.AlphaLine(px[1] + 1));
  m_focalmap->SetDetector(m_detectmap->DetectorSample(), m_detectmap->DetectorLine());
  m_distortmap->SetFocalPlane(m_focalmap->FocalPlaneX(), m_focalmap->FocalPlaneY());
  
  std::vector<double> look(3);
  look[0] = m_distortmap->UndistortedFocalPlaneX();
  look[1] = m_distortmap->UndistortedFocalPlaneY();
  look[2] = m_distortmap->UndistortedFocalPlaneZ();
  VectorProxy<double,3> result(&look[0]);
  result = normalize(result);
  look = m_camera->instrumentRotation()->J2000Vector(look);
  look = m_camera->bodyRotation()->ReferenceVector(look);
  return result;
}

Vector3
IsisInterfaceFrame::camera_center(Vector2 const& /*pix*/) const {
  return m_center;
}

Quat
IsisInterfaceFrame::camera_pose(Vector2 const& /*pix*/) const {
  return m_pose;
}
