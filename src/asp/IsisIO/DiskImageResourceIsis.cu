#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file DiskImageResourceDDD.cc
///
#ifdef _MSC_VER
#pragma warning(disable:4244)
#pragma warning(disable:4267)
#pragma warning(disable:4996)
#endif

#include <vw/Core/Exception.h>
#include <vw/Math/BBox.h>
#include <vw/Math/Vector.h>
#include <vw/Image/ImageResource.h>
#include <vw/Image/PixelTypeInfo.h>
#include <asp/IsisIO/DiskImageResourceIsis.h>

#include <string>

#include <Cube.h>
#include <Portal.h>
#include <SpecialPixel.h>

using namespace std;
using namespace boost;

namespace vw {

  // We use a fixed tile size of 2048x2048 pixels here.  Although this
  // may not be the native tile size of the ISIS cube, it seems to be
  // much faster to let the ISIS driver aggregate smaller blocks by
  // making a larger request rather than caching those blocks ourselves.
  Vector2i DiskImageResourceIsis::block_read_size() const {
    return Vector2i(2048,2048);
  }

  /// Bind the resource to a file for writing.
  void DiskImageResourceIsis::create(std::string const& /*filename*/,
                                     ImageFormat const& /*format*/)
  {
    throw NoImplErr() << "The Isis driver does not yet support creation of Isis files";
  }

  /// Bind the resource to a file for reading.  Confirm that we can open
  /// the file and that it has a sane pixel format.
  void DiskImageResourceIsis::open(std::string const& filename) {
    m_cube     = boost::shared_ptr<Isis::Cube>(new Isis::Cube());
    m_filename = filename;
    m_cube->open(QString::fromStdString(m_filename));
    VW_ASSERT(m_cube->isOpen(), IOErr() << "DiskImageResourceIsis: Could not open cube file: \"" << filename << "\".");

    // Code copied from DiskImageResourceGDAL.cc, with some modifications,
    // to fix a crash in loading an ISIS .cub file with 4 bands.
    if ( m_cube->bandCount() == 1) {
      m_format.pixel_format = VW_PIXEL_GRAY;
      m_format.planes = 1;
    } else if (m_cube->bandCount() == 2) {
      m_format.pixel_format = VW_PIXEL_GRAYA;
      m_format.planes = 1;
    } else if (m_cube->bandCount() == 3) {
      m_format.pixel_format = VW_PIXEL_RGB;
      m_format.planes = 1;
    } else if (m_cube->bandCount() == 4) {
      m_format.pixel_format = VW_PIXEL_RGBA;
      m_format.planes = 1;
    } else {
      m_format.pixel_format = VW_PIXEL_SCALAR;
      m_format.planes = m_cube->bandCount();
    }
    
    // Extract the dimensions of the image
    m_format.cols = m_cube->sampleCount();
    m_format.rows = m_cube->lineCount();

    // Set member variables according to the specified pixel type
    Isis::PixelType isis_ptype = m_cube->pixelType();
    switch (isis_ptype) {
      case Isis::UnsignedByte:    m_bytes_per_pixel = 1;  m_format.channel_type = VW_CHANNEL_UINT8;    break;
      case Isis::SignedByte:      m_bytes_per_pixel = 1;  m_format.channel_type = VW_CHANNEL_INT8;     break;
      case Isis::UnsignedWord:    m_bytes_per_pixel = 2;  m_format.channel_type = VW_CHANNEL_UINT16;   break;
      case Isis::SignedWord:      m_bytes_per_pixel = 2;  m_format.channel_type = VW_CHANNEL_INT16;    break;
      case Isis::UnsignedInteger: m_bytes_per_pixel = 4;  m_format.channel_type = VW_CHANNEL_UINT32;   break;
      case Isis::SignedInteger:   m_bytes_per_pixel = 4;  m_format.channel_type = VW_CHANNEL_INT32;    break;
      case Isis::Real:            m_bytes_per_pixel = 4;  m_format.channel_type = VW_CHANNEL_FLOAT32;  break;
      case Isis::Double:          m_bytes_per_pixel = 8;  m_format.channel_type = VW_CHANNEL_FLOAT64;  break;
      default:
        vw_throw(IOErr() << "DiskImageResourceIsis: Unknown pixel type.");
    }
  }

  /// Read the disk image into the given buffer.
  void DiskImageResourceIsis::read(ImageBuffer const& dest, BBox2i const& bbox) const {
    VW_ASSERT(bbox.max().x() <= m_cube->sampleCount() ||
              bbox.max().y() <= m_cube->lineCount(),
              IOErr() << "DiskImageResourceIsis: requested bbox " << bbox
              << " exceeds image dimensions [" << m_cube->sampleCount()
              << " " << m_cube->lineCount() << "]");

    // Read in the requested tile from the cube file.  Note that ISIS
    // cube pixel indices appear to be 1-based.
    Isis::Portal buffer(bbox.width(), bbox.height(), m_cube->pixelType());
    buffer.SetPosition(bbox.min().x()+1, bbox.min().y()+1, 1);
    m_cube->read(buffer);

    // Create generic image buffer from the Isis data.
    ImageBuffer src;
    src.data = buffer.RawBuffer();
    src.format = m_format;
    src.format.cols = bbox.width();
    src.format.rows = bbox.height();
    src.cstride = m_bytes_per_pixel;
    src.rstride = m_bytes_per_pixel * bbox.width();
    src.pstride = m_bytes_per_pixel * bbox.width() * bbox.height();

    convert(dest, src);
  }

  // Write the given buffer into the disk image.
  void DiskImageResourceIsis::write(ImageBuffer const& /*src*/, BBox2i const& /*bbox*/) {
    vw_throw ( NoImplErr() << "The Isis driver does not yet support creation of Isis files" );
  }

  // A FileIO hook to open a file for reading
  DiskImageResource* DiskImageResourceIsis::construct_open(std::string const& filename) {
    return new DiskImageResourceIsis(filename);
  }

  // A FileIO hook to open a file for writing
  DiskImageResource*
  DiskImageResourceIsis::construct_create(std::string const& filename,
                                          ImageFormat const& format)
  {
    return new DiskImageResourceIsis(filename, format);
  }

  /// Info about special pixel types in ISIS
  //  --------------------------------------
  double DiskImageResourceIsis::nodata_read() const {
    switch (m_format.channel_type) {
      case VW_CHANNEL_FLOAT64: return Isis::NULL8;
      case VW_CHANNEL_FLOAT32: return Isis::NULL4;
      case VW_CHANNEL_INT32:   return Isis::INULL4;
      case VW_CHANNEL_INT16:   return Isis::NULL2;
      default:                 return 0.0;
    }
  }
  double DiskImageResourceIsis::valid_minimum() const {
    switch (m_format.channel_type) {
      case VW_CHANNEL_FLOAT64: return Isis::ValidMinimum;
      case VW_CHANNEL_FLOAT32: return Isis::VALID_MIN4;
      case VW_CHANNEL_INT32:   return Isis::IVALID_MIN4;
      case VW_CHANNEL_INT16:   return Isis::VALID_MIN2;
      case VW_CHANNEL_UINT16:  return Isis::VALID_MINU2;
      default:                 return Isis::VALID_MIN1;
    }
  }
  double DiskImageResourceIsis::valid_maximum() const {
    switch (m_format.channel_type) {
      case VW_CHANNEL_FLOAT64: return Isis::ValidMaximum;
      case VW_CHANNEL_FLOAT32: return Isis::VALID_MAX4;
      case VW_CHANNEL_INT32:   return 2147483647;
      case VW_CHANNEL_INT16:   return Isis::VALID_MAX2;
      case VW_CHANNEL_UINT16:  return Isis::VALID_MAXU2;
      default:                 return Isis::VALID_MAX1;
    }
  }
  bool DiskImageResourceIsis::is_map_projected() const {
    // They used to have a HasProjection. I'm not sure if this fix is the correct method now.
    return m_cube->projection() != NULL;
  }
}
