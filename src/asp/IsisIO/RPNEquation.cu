#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Core/Exception.h>
#include <vw/Math/Vector.h>
#include <asp/IsisIO/RPNEquation.h>

#include <iomanip>
#include <stack>
#include <vector>

#include <boost/algorithm/string/classification.hpp>
#include <boost/algorithm/string/split.hpp>

using namespace vw;
using namespace asp;

// Constructors
//-----------------------------------------------------
RPNEquation::RPNEquation() {
  m_x_eq.clear();
  m_x_consts.clear();
  m_y_eq.clear();
  m_y_consts.clear();
  m_z_eq.clear();
  m_z_consts.clear();
  m_cached_time = -1;
  m_time_offset = 0;
}
RPNEquation::RPNEquation( std::string const& x_eq,
                          std::string const& y_eq,
                          std::string const& z_eq ) {
  string_to_eqn( x_eq, m_x_eq, m_x_consts );
  string_to_eqn( y_eq, m_y_eq, m_y_consts );
  string_to_eqn( z_eq, m_z_eq, m_z_consts );
  m_cached_time = -1;
  m_time_offset = 0;
}

// Update
//-----------------------------------------------------
void RPNEquation::update( double t ) {
  m_cached_time = t;
  double delta_t = t - m_time_offset;
  m_cached_output[0] = evaluate( m_x_eq,
                                 m_x_consts,
                                 delta_t );
  m_cached_output[1] = evaluate( m_y_eq,
                                 m_y_consts,
                                 delta_t );
  m_cached_output[2] = evaluate( m_z_eq,
                                 m_z_consts,
                                 delta_t );
}
void RPNEquation::string_to_eqn( std::string const& str,
                                 std::vector<std::string>& commands,
                                 std::vector<double>& consts ) {
  // Breaks a string into the equation format used internally
  commands.clear();
  consts.clear();
  boost::split( commands, str, boost::is_any_of(" ="));

  // Cleaning out any tokens that are just ""
  for(std::vector<std::string>::iterator iter = commands.begin();
      iter != commands.end(); ++iter ) {
    if ( (*iter) == "" ) {
      iter = commands.erase(iter);
      iter--;
    }
  }

  // Pulling out the numbers
  for(std::vector<std::string>::iterator iter = commands.begin();
      iter != commands.end(); ++iter ) {
    if ( isdigit( (*iter)[(*iter).size()-1] ) ) {
      consts.push_back( atof( iter->c_str() ) );
      *iter = "c";
    }
  }
}

double RPNEquation::evaluate( std::vector<std::string> const& commands,
                              std::vector<double>& consts,
                              double t ) {
  // Evaluates an equation in the internal format
  if ( commands.empty() )
    return 0;
  int consts_index = 0;
  std::stack<double> rpn_stack;
  double buffer;
  for ( std::vector<std::string>::const_iterator iter = commands.begin();
        iter != commands.end(); ++iter ) {
    if ( *iter == "c" ) {
      rpn_stack.push( consts[consts_index] );
      consts_index++;
    } else if ( *iter == "t" ) {
      rpn_stack.push( t );
    } else if ( rpn_stack.size() < 1 ) {
      vw_throw( IOErr() << "Insufficient arguments for RPN command: "
                << *iter << "\n" );
    } else if ( *iter == "sin" ) {
      buffer = sin( rpn_stack.top() );
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( *iter == "cos" ) {
      buffer = cos( rpn_stack.top() );
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( *iter == "tan" ) {
      buffer = tan( rpn_stack.top() );
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( *iter == "abs" ) {
      buffer = fabs( rpn_stack.top() );
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( rpn_stack.size() < 2 ) {
      vw_throw( IOErr() << "Insufficient arguments for command: "
                << *iter << "\n" );
    } else if ( *iter == "*" ) {
      buffer = rpn_stack.top();
      rpn_stack.pop();
      buffer *= rpn_stack.top();
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( *iter == "/" ) {
      buffer = rpn_stack.top();
      rpn_stack.pop();
      buffer = rpn_stack.top() / buffer;
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( *iter == "-" ) {
      buffer = rpn_stack.top();
      rpn_stack.pop();
      buffer = rpn_stack.top() - buffer;
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( *iter == "+" ) {
      buffer = rpn_stack.top();
      rpn_stack.pop();
      buffer += rpn_stack.top();
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else if ( *iter == "^" ) {
      buffer = rpn_stack.top();
      rpn_stack.pop();
      buffer = pow( rpn_stack.top(), buffer );
      rpn_stack.pop();
      rpn_stack.push( buffer );
    } else {
      vw_throw( IOErr() << "Unknown RPN operator: " << *iter << "\n" );
    }
  } // End of calculator

  if ( rpn_stack.size() != 1 )
    vw_throw( IOErr() << "Unbalanced RPN equation! More constants than need by operators.\n" );

  return rpn_stack.top();
}

// FileIO
//-----------------------------------------------------
void RPNEquation::write( std::ofstream &f ) {
  for ( int i = 0; i < 3; i++ ) {
    std::vector<std::string>* eq_ptr = NULL;
    std::vector<double>* cs_ptr = NULL;
    switch(i) {
    case 0:
      eq_ptr = &m_x_eq;
      cs_ptr = &m_x_consts;
      break;
    case 1:
      eq_ptr = &m_y_eq;
      cs_ptr = &m_y_consts;
      break;
    case 2:
      eq_ptr = &m_z_eq;
      cs_ptr = &m_z_consts;
      break;
    }

    f << std::setprecision( 15 );
    int cs_idx = 0;
    for ( unsigned j = 0; j < eq_ptr->size(); j++ ) {
      if ( (*eq_ptr)[j] == "c" ) {
        f << (*cs_ptr)[cs_idx] << " ";
        cs_idx++;
      } else {
        f << (*eq_ptr)[j] << " ";
      }
    }
    f << "\n";
  }
}

void RPNEquation::read( std::ifstream &f ) {
  std::string buffer;
  m_cached_time = -1;

  buffer.clear();
  std::getline( f, buffer );
  string_to_eqn( buffer, m_x_eq, m_x_consts );

  buffer.clear();
  std::getline( f, buffer );
  string_to_eqn( buffer, m_y_eq, m_y_consts );

  buffer.clear();
  std::getline( f, buffer );
  string_to_eqn( buffer, m_z_eq, m_z_consts );
}

// Constant Access
//-----------------------------------------------------
double& RPNEquation::operator[]( size_t n ) {
  m_cached_time = -1;
  if ( n >= m_x_consts.size() + m_y_consts.size()
       + m_z_consts.size() )
    vw_throw( ArgumentErr() << "RPNEquation: invalid index." );
  if ( n < m_x_consts.size() )
    return m_x_consts[n];
  else if ( n < m_x_consts.size() + m_y_consts.size() )
    return m_y_consts[n-m_x_consts.size()];
  return m_z_consts[n-m_x_consts.size()-m_y_consts.size()];
}
