#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Core/Common.h>
#include <asp/Core/CameraTransforms.h>
#include <asp/Camera/CsmModel.h>
#include <asp/Camera/SyntheticLinescan.h>
#include <asp/Camera/SatSim.h>

#include <usgscsm/UsgsAstroLsSensorModel.h>

#include <vw/Camera/PinholeModel.h>
#include <vw/Image/ImageViewRef.h>
#include <vw/Cartography/CameraBBox.h>
#include <vw/Math/Functors.h>
#include <vw/Core/Stopwatch.h>

namespace asp {

// Populate the CSM model with the given camera positions and orientations. Note
// that opt.num_cameras is the number of cameras within the desired orbital segment
// of length orbit_len. We will have extra cameras beyond that segment to make it 
// easy to interpolate the camera position and orientation at any time and also to 
// solve for jitter. We can have -opt.num_cams/2 <= i < 2*opt.num_cams - opt.num_cams/2. 
// When 0 <= i < opt.num_cams, we are within the orbital segment.
void populateSyntheticLinescan(SatSimOptions const& opt, 
                      double orbit_len, 
                      vw::cartography::GeoReference const & georef,    
                      std::map<int, vw::Vector3>    const & positions,
                      std::map<int, vw::Matrix3x3>  const & cam2world,
                      // Outputs
                      asp::CsmModel & model) {

  // Must have as many positions as orientations
  if (positions.size() != cam2world.size())
    vw_throw(vw::ArgumentErr() << "Expecting as many positions as orientations.\n");

  // Do not use a precision below 1.0-e8 as then the linescan model will return junk.
  model.m_desired_precision = asp::DEFAULT_CSM_DESIRED_PRECISISON;
  model.m_semi_major_axis = georef.datum().semi_major_axis();
  model.m_semi_minor_axis = georef.datum().semi_minor_axis();

  // Create the linescan model. Memory is managed by m_gm_model.
  model.m_gm_model.reset(new UsgsAstroLsSensorModel);
  UsgsAstroLsSensorModel* ls_model
    = dynamic_cast<UsgsAstroLsSensorModel*>(model.m_gm_model.get());
  if (ls_model == NULL)
    vw::vw_throw(vw::ArgumentErr() << "Invalid initialization of the linescan model.\n");

  // This performs many initializations apart from the above
  ls_model->reset();

  // Override some initializations
  ls_model->m_nSamples         = opt.image_size[0]; 
  ls_model->m_nLines           = opt.image_size[1];
  ls_model->m_platformFlag     = 1; // Use 1, for order 8 Lagrange interpolation
  ls_model->m_minElevation     = -10000.0; // -10 km
  ls_model->m_maxElevation     =  10000.0; //  10 km
  ls_model->m_focalLength      = opt.focal_length;
  ls_model->m_zDirection       = 1.0;
  ls_model->m_halfSwath        = 1.0;
  ls_model->m_sensorIdentifier = "SyntheticLinescan";
  ls_model->m_majorAxis        = model.m_semi_major_axis;
  ls_model->m_minorAxis        = model.m_semi_minor_axis;
  
  // The choices below are copied from the DigitalGlobe CSM linescan model.
  // Better to keep same convention than dig deep inside UsAstroLsSensorModel.
  // Also keep in mind that a CSM pixel has extra 0.5 added to it.
  vw::Vector2 detector_origin;
  detector_origin[0]                 = -opt.optical_center[0]; 
  detector_origin[1]                 = 0.0;
  ls_model->m_iTransL[0]             = 0.0;  
  ls_model->m_iTransL[1]             = 0.0;
  ls_model->m_iTransL[2]             = 1.0;
  ls_model->m_iTransS[0]             = 0.0;
  ls_model->m_iTransS[1]             = 1.0;
  ls_model->m_iTransS[2]             = 0.0;
  ls_model->m_detectorLineOrigin     = 0.0;
  ls_model->m_detectorSampleOrigin   = 0.0;
  ls_model->m_detectorLineSumming    = 1.0;
  ls_model->m_startingDetectorLine   = detector_origin[1];
  ls_model->m_detectorSampleSumming  = 1.0;
  ls_model->m_startingDetectorSample = (detector_origin[0] - 0.5);

  // Set the time. The first image line time is 0. The last image line time
  // will depend on distance traveled and speed.
  double beg_t = 0.0;
  double end_t = orbit_len / opt.velocity;
  double dt = (end_t - beg_t) / (opt.image_size[1] - 1.0);
  ls_model->m_intTimeLines.push_back(1.0); // to offset CSM's quirky 0.5 additions in places
  ls_model->m_intTimeStartTimes.push_back(beg_t);
  ls_model->m_intTimes.push_back(dt);

  // Positions and velocities. Note how, as above, there are more positions than
  // opt.num_cameras as they extend beyond orbital segment. So care is needed
  // below. Time is 0 when we reach the first image line, and it is end_t at the
  // last line. Positions before that have negative time. Time at position with
  // index i is m_t0Ephem + i*m_dtEphem, if index 0 is for the earliest postion,
  // but that is way before the orbital segment starting point which is the
  // first image line. We can have -opt.num_cams/2 <= pos_it->first <
  // 2*opt.num_cams - opt.num_cams/2.
  int beg_pos_index = positions.begin()->first; // normally equals -opt.num_cameras/2
  if (beg_pos_index > 0)
    vw::vw_throw(vw::ArgumentErr() << "First position index must be non-positive.\n");
  ls_model->m_numPositions = 3 * positions.size(); // concatenate all coordinates
  ls_model->m_dtEphem = (end_t - beg_t) / (opt.num_cameras - 1.0); // care here
  ls_model->m_t0Ephem = beg_t + beg_pos_index * ls_model->m_dtEphem; // care here

  ls_model->m_positions.resize(ls_model->m_numPositions);
  ls_model->m_velocities.resize(ls_model->m_numPositions);
  for (auto pos_it = positions.begin(); pos_it != positions.end(); pos_it++) {
    int index = pos_it->first - beg_pos_index; // so we can start at 0
    auto ctr = pos_it->second;
    for (int coord = 0; coord < 3; coord++) {
      ls_model->m_positions [3*index + coord] = ctr[coord];
      ls_model->m_velocities[3*index + coord] = 0.0; // should not be used
    }
  }

  // Orientations. Care with defining dt as above.
  int beg_quat_index = cam2world.begin()->first; // normally equals -opt.num_cameras/2
  if (beg_quat_index > 0)
    vw::vw_throw(vw::ArgumentErr() << "First orientation index must be non-positive.\n");
  if (beg_pos_index != beg_quat_index)
    vw::vw_throw(vw::ArgumentErr() 
      << "First position index must equal first orientation index.\n");
      
  ls_model->m_numQuaternions = 4 * cam2world.size();
  ls_model->m_dtQuat = (end_t - beg_t) / (opt.num_cameras - 1.0);
  ls_model->m_t0Quat = beg_t + beg_quat_index * ls_model->m_dtQuat;

  ls_model->m_quaternions.resize(ls_model->m_numQuaternions);
  for (auto quat_it = cam2world.begin(); quat_it != cam2world.end(); quat_it++) {
    int index = quat_it->first - beg_quat_index; // so we can start at 0

    // Find the quaternion at this index.
    auto c2w = quat_it->second;
    double x, y, z, w;
    asp::matrixToQuaternion(c2w, x, y, z, w);

    // Note how we store the quaternions in the order x, y, z, w, not w, x, y, z.
    int coord = 0;
    ls_model->m_quaternions[4*index + coord] = x; coord++;
    ls_model->m_quaternions[4*index + coord] = y; coord++;
    ls_model->m_quaternions[4*index + coord] = z; coord++;
    ls_model->m_quaternions[4*index + coord] = w; coord++;
  }

  // Re-creating the model from the state forces some operations to
  // take place which are inaccessible otherwise.
  std::string modelState = ls_model->getModelState();
  ls_model->replaceModelState(modelState);
}

// Allow finding the time at any line, even negative ones. Here a
// simple slope-intercept formula is used rather than a table. 
// This was a temporary function used for debugging
// double get_time_at_line(double line) const {
//     csm::ImageCoord csm_pix;
//     asp::toCsmPixel(vw::Vector2(0, line), csm_pix);
//     return ls_model->getImageTime(csm_pix);
// }

// The pointing vector in sensor coordinates, before applying cam2world. This
// is for testing purposes. Normally CSM takes care of this internally.
// This was a temporary function used for debugging
// vw::Vector3 get_local_pixel_to_vector(vw::Vector2 const& pix) const {

//   vw::Vector3 result(pix[0] + detector_origin[0], 
//                       detector_origin[1], 
//                       ls_model->m_focalLength);
//   // Make the direction have unit length
//   result = normalize(result);
//   return result;
// }

// Compare the camera center and direction with pinhole. A very useful
// test.
void PinLinescanTest(SatSimOptions                const & opt, 
                     asp::CsmModel                const & ls_cam,
                     std::map<int, vw::Vector3>   const & positions,
                     std::map<int, vw::Matrix3x3> const & cam2world) {
                        
  for (int i = 0; i < int(positions.size()); i++) {

    auto pin_cam 
      = vw::camera::PinholeModel(asp::mapVal(positions, i),
                                 asp::mapVal(cam2world, i),
                                 opt.focal_length, opt.focal_length,
                                 opt.optical_center[0], opt.optical_center[1]);
  
    double line = (opt.image_size[1] - 1.0) * i / std::max((positions.size() - 1.0), 1.0);
  
    // Need care here
    vw::Vector2 pin_pix(opt.optical_center[0], opt.optical_center[1]);
    vw::Vector2 ls_pix (opt.optical_center[0], line);

    // The differences below must be 0
    vw::Vector3 ls_ctr  = ls_cam.camera_center(ls_pix);
    vw::Vector3 pin_ctr = pin_cam.camera_center(pin_pix);
    std::cout << "ls ctr and and pin - ls ctr diff: " << ls_ctr << " "
              << norm_2(pin_ctr - ls_ctr) << std::endl;

    vw::Vector3 ls_dir = ls_cam.pixel_to_vector(ls_pix);
    vw::Vector3 pin_dir = pin_cam.pixel_to_vector(pin_pix);
    std::cout << "ls dir and pin - ls dir diff: " << ls_dir << " "
              << norm_2(pin_dir - ls_dir) << std::endl;
  }
}

// Wrapper for logic to intersect DEM with ground. The xyz provided on input serves
// as initial guess and gets updated on output if the intersection succeeds. Return
// true on success.
bool intersectDemWithRay(SatSimOptions const& opt,
                         vw::cartography::GeoReference const& dem_georef,vw::ImageViewRef<vw::PixelMask<float>> dem,
                         vw::Vector3 const& cam_ctr, 
                         vw::Vector3 const& cam_dir,
                         double height_guess,
                         // Output
                         vw::Vector3 & xyz) {

    // Find the intersection of this ray with the ground
    bool treat_nodata_as_zero = false;
    bool has_intersection = false;
    double max_abs_tol = std::min(opt.dem_height_error_tol, 1e-14);
    double max_rel_tol = max_abs_tol;
    int num_max_iter = 100;

    vw::Vector3 local_xyz 
      = vw::cartography::camera_pixel_to_dem_xyz
        (cam_ctr, cam_dir, dem, dem_georef, treat_nodata_as_zero, has_intersection, 
        // Below we use a prudent approach. Try to make the solver work
        // hard. It is not clear if this is needed.
        std::min(opt.dem_height_error_tol, 1e-8),
        max_abs_tol, max_rel_tol, 
        num_max_iter, xyz, height_guess);

    if (!has_intersection)
      return false;

    // Update xyz with produced value if we succeeded
    xyz = local_xyz;
    return true;
}

// Estimate pixel aspect ratio (width / height) of a pixel on the ground
double pixelAspectRatio(SatSimOptions                 const & opt,     
                        vw::cartography::GeoReference const & dem_georef,
                        asp::CsmModel                 const & ls_cam,
                        vw::ImageViewRef<vw::PixelMask<float>>  dem,  
                        double height_guess) {

  // Put here a stop watch
  //vw::Stopwatch sw;
  //sw.start();

  // We checked that the image width and height is at least 2 pixels. That is
  // needed to properly create the CSM model. Now do some samples to see how the
  // pixel width and height are on the ground. Use a small set of samples. Should be good
  // enough. Note how we go a little beyond each sample, while still not exceeding
  // the designed image size. 
  double samp_x = (opt.image_size[0] - 1.0) / 10.0;
  double samp_y = (opt.image_size[1] - 1.0) / 10.0;

  std::vector<double> ratios; 
  vw::Vector3 xyz(0, 0, 0); // intersection with DEM, will be updated below
  
  for (double x = 0; x < opt.image_size[0] - 1.0; x += samp_x) {
    for (double y = 0; y < opt.image_size[1] - 1.0; y += samp_y) {

      // Find the intersection of the ray from this pixel with the ground
      vw::Vector2 pix(x, y);
      vw::Vector3 ctr = ls_cam.camera_center(pix);
      vw::Vector3 dir = ls_cam.pixel_to_vector(pix);
      bool ans = intersectDemWithRay(opt, dem_georef, dem, ctr, dir, 
         height_guess, xyz);
      if (!ans) 
        continue;
      vw::Vector3 P0 = xyz;

      // Add a little to the pixel, but stay within the image bounds
      double dx = std::min(samp_x, 0.5);
      double dy = std::min(samp_y, 0.5);

      // See pixel width on the ground
      pix = vw::Vector2(x + dx, y);
      ctr = ls_cam.camera_center(pix);
      dir = ls_cam.pixel_to_vector(pix);
      ans = intersectDemWithRay(opt, dem_georef, dem, ctr, dir, 
         height_guess, xyz);
      if (!ans) 
        continue;
      vw::Vector3 Px = xyz;

      // See pixel height on the ground
      pix = vw::Vector2(x, y + dy);
      ctr = ls_cam.camera_center(pix);
      dir = ls_cam.pixel_to_vector(pix);
      ans = intersectDemWithRay(opt, dem_georef, dem, ctr, dir, 
         height_guess, xyz);
      if (!ans)
        continue;
      vw::Vector3 Py = xyz;

      double ratio = norm_2(Px - P0) / norm_2(Py - P0);
      if (std::isnan(ratio) || std::isinf(ratio) || ratio <= 0.0)
        continue;
      ratios.push_back(ratio);
    }
  }

  if (ratios.empty())
    vw::vw_throw(vw::ArgumentErr() << "No valid samples found to compute "
             << "the pixel width and height on the ground.\n");

  double ratio = vw::math::destructive_median(ratios);

  //sw.stop();
  //std::cout << "Time to compute pixel aspect ratio: " << sw.elapsed_seconds() << std::endl;

  return ratio;
}

// Create and save a linescan camera with given camera positions and orientations.
// There will be just one of them, as all poses are part of the same linescan camera.
void genLinescanCameras(double                                orbit_len, 
                        vw::cartography::GeoReference const & dem_georef,
                        vw::ImageViewRef<vw::PixelMask<float>> dem,  
                        std::map<int, vw::Vector3>    const & positions,
                        std::map<int, vw::Matrix3x3>  const & cam2world,
                        std::map<int, vw::Matrix3x3>  const & cam2world_no_jitter,
                        std::map<int, vw::Matrix3x3>  const & ref_cam2world,
                        double                                height_guess,
                        // Outputs
                        SatSimOptions                         & opt, 
                        std::vector<std::string>              & cam_names,
                        std::vector<vw::CamPtr>               & cams) {

  // Sanity checks
  if (cam2world.size() != positions.size() || cam2world_no_jitter.size() != positions.size())
    vw::vw_throw(vw::ArgumentErr() << "Expecting as many camera orientations as positions.\n");

  // Initialize the outputs
  cam_names.clear();
  cams.clear();

  // Create the camera. Will be later owned by a smart pointer.
  asp::CsmModel * ls_cam = new asp::CsmModel;

  // If creating square pixels, must use the camera without jitter to estimate
  // the image height. Otherwise the image height produced from the camera with
  // jitter will be inconsistent with the one without jitter. This is a bugfix. 
  if (!opt.square_pixels) 
    populateSyntheticLinescan(opt, orbit_len, dem_georef, positions, cam2world, 
      *ls_cam); // output 
  else
    populateSyntheticLinescan(opt, orbit_len, dem_georef, positions, cam2world_no_jitter,   
      *ls_cam); // output

  // Sanity check (very useful)
  // PinLinescanTest(opt, *ls_cam, positions, cam2world);

  if (opt.square_pixels) {
    // Find the pixel aspect ratio on the ground (x/y)
    vw::vw_out() << "Adjusting image height from " << opt.image_size[1] << " to ";
    double ratio = pixelAspectRatio(opt, dem_georef, *ls_cam, dem, height_guess);
    // Adjust the image height to make the pixels square
    opt.image_size[1] = std::max(round(opt.image_size[1] / ratio), 2.0);
    vw::vw_out() << opt.image_size[1] << " pixels, to make the ground "
                 << "projection of an image pixel be roughly square.\n";

    // Recreate the camera with this aspect ratio. This time potentially use the 
    // camera with jitter. 
    populateSyntheticLinescan(opt, orbit_len, dem_georef, positions, cam2world, *ls_cam); 
    // Sanity check (very useful for testing, the new ratio must be close to 1.0)
    // ratio = pixelAspectRatio(opt, dem_georef, *ls_cam, dem, height_guess);
  }
  std::string filename = opt.out_prefix + ".json";
  ls_cam->saveState(filename);

  if (opt.save_ref_cams) {
      asp::CsmModel ref_cam;
      populateSyntheticLinescan(opt, orbit_len, dem_georef, positions, ref_cam2world,
        ref_cam); // output
    std::string ref_filename = opt.out_prefix + "-ref.json";
    ref_cam.saveState(ref_filename);
  }

  // Save the camera name and smart pointer to camera
  cam_names.push_back(filename);
  cams.push_back(vw::CamPtr(ls_cam));

  return;
}

// A function to read Linescan cameras from disk. There will
// be just one of them, but same convention is kept as for Pinhole
// where there is many of them. Note that the camera is created as of CSM type,
// rather than asp::CsmModel type. This is not important as we will
// abstract it right away to the base class.
void readLinescanCameras(SatSimOptions const& opt, 
    std::vector<std::string> & cam_names,
    std::vector<vw::CamPtr> & cams) {

  // Read the camera names
  vw::vw_out() << "Reading: " << opt.camera_list << std::endl;
  asp::read_list(opt.camera_list, cam_names);

  // Sanity checks
  if (cam_names.empty())
    vw::vw_throw(vw::ArgumentErr() << "No cameras were found.\n");
  if (cam_names.size() != 1)
    vw::vw_throw(vw::ArgumentErr() << "Only one linescan camera is expected.\n");

  cams.resize(cam_names.size());
  for (int i = 0; i < int(cam_names.size()); i++)
    cams[i] = vw::CamPtr(new asp::CsmModel(cam_names[i]));

  return;
}

} // end namespace asp

