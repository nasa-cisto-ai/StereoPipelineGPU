#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Functions used for handling CSM camera models.

#include <asp/Camera/CsmModel.h>
#include <asp/Camera/CsmUtils.h>
#include <asp/Camera/JitterSolveUtils.h>
#include <vw/Cartography/GeoReference.h>
#include <vw/Cartography/GeoReferenceUtils.h>

#include <usgscsm/Utilities.h>

namespace asp {

// Normalize quaternions in UsgsAstroLsSensorModel.
void normalizeQuaternions(UsgsAstroLsSensorModel * ls_model) {

  for (int qit = 0; qit < ls_model->m_numQuaternions / 4; qit++) {

    double norm = 0.0;
    for (int coord = 0; coord < 4; coord++)
      norm += ls_model->m_quaternions[4 * qit + coord] * ls_model->m_quaternions[4 * qit + coord];

    norm = sqrt(norm);
    if (norm == 0)
      continue;
   
    for (int coord = 0; coord < 4; coord++)
      ls_model->m_quaternions[4 * qit + coord] /= norm;
  }

  return;
}

// Normalize quaternions in UsgsAstroFrameSensorModel.
void normalizeQuaternions(UsgsAstroFrameSensorModel * frame_model) {

  // Fetch the quaternions. In the model, the positions are stored first, then
  // the quaternions. 
  double q[4];
  double norm = 0.0;
  for (size_t i = 0; i < 4; i++) {
    q[i] = frame_model->getParameterValue(i + 3); 
    norm += q[i]*q[i];
  }
  norm = sqrt(norm);

  // Normalize the quaternions. Put them back in the model.
  for (size_t i = 0; i < 4; i++) {
    q[i] /= norm;
    frame_model->setParameterValue(i + 3, q[i]);
  }

  return;
}

// Get quaternions. This duplicates the UsgsAstroLsSensorModel function as that one is private
// TODO(oalexan1): Call this from LinescanDGModel.cc.
void interpQuaternions(UsgsAstroLsSensorModel * ls_model, double time,
                      double q[4]) {
  int nOrder = 8;
  if (ls_model->m_platformFlag == 0)
    nOrder = 4;
  int nOrderQuat = nOrder;
  if (ls_model->m_numQuaternions/4 < 6 && nOrder == 8)
    nOrderQuat = 4;
  
  lagrangeInterp(ls_model->m_numQuaternions / 4, &ls_model->m_quaternions[0],
                 ls_model->m_t0Quat, ls_model->m_dtQuat, time, 4, nOrderQuat, q);
}

// Get positions. Based on the UsgsAstroLsSensorModel code.
void interpPositions(std::vector<double> const& positions,
                     double t0Ephem, double dtEphem, int platformFlag,
                     double time, double pos[3]) {
  int nOrder = 8;
  if (platformFlag == 0)
    nOrder = 4;
  
  // This seems to handle gracefully when the number of positions is very small.
  lagrangeInterp(positions.size() / 3, &positions[0],
                 t0Ephem, dtEphem,
                 time, 3, nOrder, pos);
}

// Get positions. Based on the UsgsAstroLsSensorModel code.
void interpPositions(UsgsAstroLsSensorModel * ls_model, double time,
                     double pos[3]) {
  // Call the above wrapper
  interpPositions(ls_model->m_positions, ls_model->m_t0Ephem, ls_model->m_dtEphem,
                  ls_model->m_platformFlag, time, pos);
}

// Get positions. Based on the UsgsAstroLsSensorModel code.
// TODO(oalexan1): Move this to a new CsmModelUtils.cc file and
void interpVelocities(UsgsAstroLsSensorModel * ls_model, double time,
                  double vel[3]) {
  int nOrder = 8;
  if (ls_model->m_platformFlag == 0)
    nOrder = 4;
  double sensPosNom[3];
  lagrangeInterp(ls_model->m_numPositions / 3, &ls_model->m_velocities[0],
                 ls_model->m_t0Ephem, ls_model->m_dtEphem,
                 time, 3, nOrder, vel);
}

// Nearest neighbor interpolation into a sequence of vectors of length
// vectorLength, stored one after another in valueArray. The result
// goes in valueVector. Analogous to lagrangeInterp() in CSM.
void nearestNeibInterp(const int &numTimes, const double *valueArray,
                       const double &startTime, const double &delTime,
                       const double &time, const int &vectorLength,
                       double *valueVector) {
  
  if (numTimes < 1)
    vw::vw_throw(vw::ArgumentErr() << "Cannot interpolate into a vector of zero length.\n");
  
  // Compute index
  int index = round((time - startTime) / delTime);
  if (index < 0) 
    index = 0;
  if (index >= numTimes)
    index = numTimes - 1;

  int start = index * vectorLength;
  for (int i = 0; i < vectorLength; i++)
    valueVector[i] = valueArray[start + i];

  return;
}

// Given two values, double t1, t2, and to points, vw::Vector3 P1, P2, at those
// values, find the value at t using linear interpolation.
vw::Vector3 linearInterp(double t1, double t2, vw::Vector3 const& P1, vw::Vector3 const& P2, 
                         double t) {
    if (t1 == t2)
      vw::vw_throw(vw::ArgumentErr() << "Expecting t1 != t2 in interpolation.\n");

  double alpha = (t - t1)/(t2 - t1);
  return P1 + alpha*(P2 - P1);
}

// - Given a vector of positions with starting time and spacing, and a new
//   starting time, spacing, and number of points, interpolate the positions at
//   the new locations. If outside the range, use linear extrapolation. For
//   that, assume the points are in ECEF, giving an orbit, so first convert to
//   projected coordinates to make the extrapolated trajectory still go around
//   the planet rather than go on a tangent. 
// - Will return the original value at points at which the new time is the same as old
//   time (within 1e-8 time tolerance).
void orbitInterpExtrap(double t0_in, double dt_in, int platformFlag,
  std::vector<double> const& positions_in, vw::cartography::GeoReference const& geo, 
  double t0_out, double dt_out, int num_out, std::vector<double> & positions_out) {

  // Wipe the output
  positions_out.clear();

  // Sanity checks
  if (positions_in.size() < 2 || num_out < 2)
    vw::vw_throw(vw::ArgumentErr() << "Expecting at least two positions in interpolation.\n");
  if (dt_in <= 0.0 || dt_out <= 0.0)
    vw::vw_throw(vw::ArgumentErr() << "Expecting positive time step in interpolation.\n");

  // May need to create new orbital points for interpolation/extrapolation. Start
  // by putting the existing ones in a map.
  std::map<double, vw::Vector3> time_to_pos;
  for (size_t i = 0; i < positions_in.size()/NUM_XYZ_PARAMS; i++) {
    double t = t0_in + i * dt_in;
    int start = i * NUM_XYZ_PARAMS;
    vw::Vector3 pos(positions_in[start + 0], positions_in[start + 1], positions_in[start + 2]);
    time_to_pos[t] = pos;
  }

  // Add left extrapolated points. Use first two positions for linear extrapolation.
  // Extrapolation is done in projected coordinates, to stay in orbit,
  // and then converted back to ECEF. Add at least 8 points to help with Lagrange
  // interpolation later.
  auto it = time_to_pos.begin();
  vw::Vector3 P0 = it->second;
  it++; 
  vw::Vector3 P1 = it->second;
  vw::Vector3 proj0 = vw::cartography::ecefToProj(geo, P0);
  vw::Vector3 proj1 = vw::cartography::ecefToProj(geo, P1);
  double t = t0_in;
  while (t + 8 * dt_in >= t0_out) {
    t -= dt_in;
    vw::Vector3 proj = linearInterp(t0_in, t0_in + dt_in, proj0, proj1, t);
    vw::Vector3 P = vw::cartography::projToEcef(geo, proj);
    time_to_pos[t] = P;
  }

  // Now do the same at the end. Use (t1, P1) for the last point, and (t0, P0) for
  // the second to last point. Extrapolate to the right of t1.
  it = time_to_pos.end(); 
  it--;
  P1 = it->second;
  double t1 = it->first;
  it--;
  double t0 = it->first;
  P0 = it->second;
  proj0 = vw::cartography::ecefToProj(geo, P0);
  proj1 = vw::cartography::ecefToProj(geo, P1);

  // Add right extrapolated points. Need to have at least 8 to be able to
  // interpolate using Lagrange.
  double t_out_end = t0_out + (num_out - 1) * dt_out;
  t = t1; 
  while (t - 8 * dt_in <= t_out_end) {
    t += dt_in;
    vw::Vector3 proj = linearInterp(t0, t1, proj0, proj1, t);
    vw::Vector3 P = vw::cartography::projToEcef(geo, proj);
    time_to_pos[t] = P;
  }

  // Put all the produced values in the same vector, in order of time
  int num_extra = time_to_pos.size();
  double t0_extra = time_to_pos.begin()->first;
  double dt_extra = dt_in;
  std::vector<double> extra_positions(num_extra * NUM_XYZ_PARAMS);
  int count = 0;
  for (auto it = time_to_pos.begin(); it != time_to_pos.end(); it++) {
    vw::Vector3 P = it->second;
    extra_positions[count*NUM_XYZ_PARAMS + 0] = P[0];
    extra_positions[count*NUM_XYZ_PARAMS + 1] = P[1];
    extra_positions[count*NUM_XYZ_PARAMS + 2] = P[2];
    count++;
  }

  // Now we have enough positions to interpolate at
  positions_out.resize(num_out * NUM_XYZ_PARAMS);
  for (int i = 0; i < num_out; i++) {
    double t = t0_out + i * dt_out;

    // If this is in the input data, just copy the value
    double i_in_float = (t - t0_in)/dt_in;
    int i_in = round(i_in_float);
    // TODO(oalexan1): The check below is sensitive to large t values
    // and to dt being on the order of 1e-8, which does not happen in practice,
    // but should be improved, somehow.
    if (i_in >= 0 && i_in < positions_in.size()/NUM_XYZ_PARAMS &&
        std::abs(t - (t0_in + i_in*dt_in)) < 1e-8) {
      positions_out[i*NUM_XYZ_PARAMS + 0] = positions_in[i_in*NUM_XYZ_PARAMS + 0];
      positions_out[i*NUM_XYZ_PARAMS + 1] = positions_in[i_in*NUM_XYZ_PARAMS + 1];
      positions_out[i*NUM_XYZ_PARAMS + 2] = positions_in[i_in*NUM_XYZ_PARAMS + 2];
      continue;
    }

    double P[3];
    interpPositions(extra_positions, t0_extra, dt_extra, platformFlag, t, P);
    positions_out[i*NUM_XYZ_PARAMS + 0] = P[0];
    positions_out[i*NUM_XYZ_PARAMS + 1] = P[1];
    positions_out[i*NUM_XYZ_PARAMS + 2] = P[2];
  }

  return;
}

// Find interpolated/extrapolated positions at all camera pose times.
// See the function being called below for more details.
void orbitInterpExtrap(UsgsAstroLsSensorModel const * ls_model,
                       vw::cartography::GeoReference const& geo,
                       std::vector<double> & positions_out) {

  orbitInterpExtrap(ls_model->m_t0Ephem, ls_model->m_dtEphem, ls_model->m_platformFlag,  
                    ls_model->m_positions,
                    geo, ls_model->m_t0Quat, ls_model->m_dtQuat, 
                    ls_model->m_quaternions.size()/NUM_QUAT_PARAMS,
                    positions_out); // output
}

} // end namespace asp
