#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file CameraResectioning.cc

#include <asp/Camera/CameraResectioning.h>

#include <opencv2/calib3d/calib3d.hpp>
#include <Eigen/Dense>

#include <string>

using namespace vw;
using namespace vw::camera;

namespace asp {

void RodriguesToRotation(Eigen::Vector3d const& vector,
                         Eigen::Matrix3d & rotation) {
  double angle = vector.norm();
  Eigen::AngleAxisd aa(angle, vector / angle);
  rotation = aa.matrix();
}
  
/// Use OpenCV to find a Pinhole camera's position and orientation
/// based on image pixels and corresponding ground positions
void findCameraPose(std::vector<vw::Vector3> const& ground_points, 
                    std::vector<vw::Vector2> const& pixel_observations,
                    vw::camera::PinholeModel & cam) {

  if (ground_points.size() != pixel_observations.size())
    vw::vw_throw(vw::ArgumentErr()
                 << "There must be as many ground points as pixel observations.\n");

  if (ground_points.size() < 4) 
    vw::vw_throw(vw::ArgumentErr()
                 << "Must have at least four points per camera to be able to "
                 << "find a camera's orientation.\n");

  Vector2 focal_length   = cam.focal_length();
  double pixel_pitch     = cam.pixel_pitch();
  Vector2 optical_offset = cam.point_offset();

  // Intrinsics
  cv::Mat intrinsics(3, 3, cv::DataType<double>::type, 0.0);
  intrinsics.at<double>(0, 0) = focal_length[0]   / pixel_pitch;
  intrinsics.at<double>(1, 1) = focal_length[1]   / pixel_pitch;
  intrinsics.at<double>(0, 2) = optical_offset[0] / pixel_pitch;
  intrinsics.at<double>(1, 2) = optical_offset[1] / pixel_pitch;
  intrinsics.at<double>(2, 2) = 1.0;

  // Assume no distortion, as that one is hard to communicate.
  // This should give a good enough initial camera. Later it will be
  // refined with bundle adjustment taking into account the distortion.
  cv::Mat distortion(4, 1, cv::DataType<double>::type, cv::Scalar(0));

  // Convert to OpenCV format
  std::vector<cv::Point2d> cv_pixel_observations;
  std::vector<cv::Point3d> cv_ground_points;
  for (size_t it = 0; it < ground_points.size(); it++) {
    auto & V = ground_points[it];
    auto & P = pixel_observations[it];
    cv_ground_points.push_back(cv::Point3d(V[0], V[1], V[2]));
    cv_pixel_observations.push_back(cv::Point2d(P[0], P[1]));
  }
  
  // Call PnP
  bool useExtrinsicGuess = false;
  int iterationsCount = 1000; // This algorithm is cheap, let it try hard
  float reprojectionError = 20.0; // because of un-modeled distortion, relax things here
  double confidence = 0.95;
  cv::Mat rvec(3, 1, cv::DataType<double>::type, cv::Scalar(0)); // Rodrigues rotation 
  cv::Mat tvec(3, 1, cv::DataType<double>::type, cv::Scalar(0)); // translation
  bool result = cv::solvePnPRansac(cv_ground_points, cv_pixel_observations,
                                   intrinsics, distortion,
                                   rvec, tvec, // outputs
                                   useExtrinsicGuess, iterationsCount, reprojectionError,
                                   confidence);
  if (!result)
    vw::vw_throw(vw::ArgumentErr()
                 << "Failed to find camera orientation using pixel and ground data.\n");

  // Convert obtained rotation
  Eigen::Matrix3d rotation;
  RodriguesToRotation(Eigen::Vector3d(rvec.at<double>(0), rvec.at<double>(1), rvec.at<double>(2)),
                      rotation);

  // Make world2cam into cam2world
  Eigen::Matrix3d cam2world = rotation.inverse(); 
  Eigen::Vector3d cam_ctr = -rotation.inverse() *
    Eigen::Vector3d(tvec.at<double>(0), tvec.at<double>(1), tvec.at<double>(2));

  // Convert Eigen matrix and vector to VW
  vw::Matrix3x3 rot;
  for (int row = 0; row < 3; row++) {
    for (int col = 0; col < 3; col++) {
      rot(row, col) = cam2world(row, col);
    }
  }
  vw::Vector3 ctr;
  for (int row = 0; row < 3; row++)
    ctr[row] = cam_ctr[row];

  // Apply the transform to the camera
  cam.set_camera_pose(rot);
  cam.set_camera_center(ctr);

  return;
}
  
} // end namespace asp
