#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/FileIO/FileUtils.h>

#include <asp/Core/StereoSettings.h>
#include <asp/Camera/CsmModel.h>

#include <boost/dll.hpp>
#include <boost/dll/runtime_symbol_info.hpp>
#include <boost/filesystem.hpp>
#include <boost/algorithm/string/case_conv.hpp>
#include <boost/version.hpp>
#include <boost/config.hpp>

// From the CSM base interface library
#include <csm/csm.h>
#include <csm/Plugin.h>
#include <csm/RasterGM.h>
#include <nlohmann/json.hpp>

// USGSCSM linescan
#include <usgscsm/UsgsAstroFrameSensorModel.h>
#include <usgscsm/UsgsAstroLsSensorModel.h>
#include <usgscsm/UsgsAstroPushFrameSensorModel.h>
#include <usgscsm/UsgsAstroSarSensorModel.h>
#include <usgscsm/Utilities.h>

#include <ale/Rotation.h>
#include <Eigen/Dense>
#include <Eigen/Geometry>

#include <streambuf>

namespace dll = boost::dll;
namespace fs = boost::filesystem;
using json = nlohmann::json;

using namespace vw;

namespace asp {

// This was discussed with the USGS folks. To convert from ISIS to ASP
// pixels we subtract 1.0. To convert from CSM pixels we have to
// subtract only 0.5.
const vw::Vector2 ASP_TO_CSM_SHIFT(0.5, 0.5);

enum USGSCSM_MODEL_TYPE {
  USGSCSM_FRAME_MODEL,
  USGSCSM_LINESCAN_MODEL,
  USGSCSM_PUSHFRAME_MODEL,
  USGSCSM_SAR_MODEL
};

vw::Mutex csm_init_mutex;

// -----------------------------------------------------------------
// Helper functions

csm::EcefCoord vectorToEcefCoord(Vector3 v) {
  csm::EcefCoord c;
  c.x = v[0];
  c.y = v[1];
  c.z = v[2];
  return c;
}

csm::ImageCoord vectorToImageCoord(Vector2 v) {
  csm::ImageCoord c;
  c.samp = v[0];
  c.line = v[1];
  return c;
}

// Auxiliary functions to convert a pixel from ASP conventions to what CSM
// expects and vice versa
void toCsmPixel(vw::Vector2 const& pix, csm::ImageCoord & csm) {
  csm.samp = pix[0] + ASP_TO_CSM_SHIFT[0];
  csm.line = pix[1] + ASP_TO_CSM_SHIFT[1];
}
void fromCsmPixel(vw::Vector2 & pix, csm::ImageCoord const& csm) {
  pix[0] = csm.samp - ASP_TO_CSM_SHIFT[0];
  pix[1] = csm.line - ASP_TO_CSM_SHIFT[1];
}


Vector3 ecefCoordToVector(csm::EcefCoord c) {
  Vector3 v;
  v[0] = c.x;
  v[1] = c.y;
  v[2] = c.z;
  return v;
}

Vector3 ecefVectorToVector(csm::EcefVector c) {
  Vector3 v;
  v[0] = c.x;
  v[1] = c.y;
  v[2] = c.z;
  return v;
}

Vector2 imageCoordToVector(csm::ImageCoord c) {
  Vector2 v;
  v[0] = c.samp;
  v[1] = c.line;
  return v;
}

// -----------------------------------------------------------------
// CsmModel class functions
CsmModel::CsmModel():m_semi_major_axis(0.0),
                     m_semi_minor_axis(0.0),
                     m_sun_position(vw::Vector3()),
                     // Do not make the precision lower than 1e-8. CSM can give
                     // junk results when it is too low.
                     m_desired_precision(asp::DEFAULT_CSM_DESIRED_PRECISISON){}
                                      
CsmModel::CsmModel(std::string const& isd_path) {
  load_model(isd_path);
}

CsmModel::~CsmModel() {
  // nothing to do.
}

bool CsmModel::file_has_isd_extension(std::string const& path) {
  std::string ext = vw::get_extension(path);
  return ((ext == ".json") || (ext == ".isd"));
}

std::string CsmModel::get_csm_plugin_folder() {

  // Look up the CSM_PLUGIN_PATH environmental variable.
  // It is set in the "libexec/libexec-funcs.sh" deploy file.
  // If the plugin is not found in CSM_PLUGIN_PATH, look at ISISROOT.
  std::string plugin_path;
  char * plugin_path_arr = getenv("CSM_PLUGIN_PATH");

  char * isis_root = getenv("ISISROOT");
  if (isis_root == NULL)
    vw_throw(vw::ArgumentErr() << "The variable ISISROOT was not set.\n");
  
  if (plugin_path_arr != NULL && std::string(plugin_path_arr) != ""){
    plugin_path = std::string(plugin_path_arr);

  }else{
    // This is for when ASP is installed without the deploy file.
    // vw_out() << "The environmental variable CSM_PLUGIN_PATH was not set.\n";
    fs::path try_path(isis_root);
    try_path /= "lib";
    plugin_path = try_path.string();
    //vw_out() << "Looking in " << plugin_path << ".\n";
  }

  if (!fs::exists(plugin_path)){
    vw_throw(ArgumentErr() << "Could not find CSM plugin folder: " << plugin_path << ".\n"
              << "Check the value of the environmental variable CSM_PLUGIN_PATH.");
  }

  return plugin_path;
}

// The original idea here was to look at every library in the plugins
// directory and load the valid plugins. For now however there is just
// one plugin, libusgscsm, and it is stored in 'lib', among thousands
// of other inapplicable libraries. Hence just pick that one.  One day
// we will have a dedicated plugins directory.
size_t CsmModel::find_csm_plugins(std::vector<std::string> &plugins) {

  plugins.clear();

  const std::string folder = get_csm_plugin_folder();

  std::string ext;
  std::vector<std::string> potential_plugins;
  std::string platform = std::string(BOOST_PLATFORM);
  boost::to_lower(platform);
  if (std::string(platform).find("linux") != std::string::npos)
    ext = ".so";
  else if (std::string(platform).find("mac") != std::string::npos) 
    ext = ".dylib";
  else
    vw_throw(ArgumentErr() << "Unknown operating system: " << BOOST_PLATFORM << "\n");

#if 0
  size_t potential_num_dlls = vw::get_files_in_folder(folder, potential_plugins, ext);
  for (size_t i = 0; i < potential_num_dlls; i++) {
    if (potential_plugins[i] != "libusgscsm" + ext) {
      continue;
    }
    
    fs::path p(folder);
    p /= potential_plugins[i];
    plugins.push_back(p.string());
  }
#endif

  fs::path p(folder);
  p /= "libusgscsm" + ext;
  std::string plugin = p.string();
  if (!fs::exists(plugin)) 
    vw_throw(ArgumentErr() << "Cannot find plugin: " <<plugin <<
              ". Set CSM_PLUGIN_PATH to the directory where the plugins are stored.\n");
  plugins.push_back(plugin);

  return plugins.size();
}

void CsmModel::print_available_models() {

  csm::PluginList available_plugins = csm::Plugin::getList();
  // vw_out() << "Detected " << available_plugins.size() << " available CSM plugin(s).\n";

  csm::PluginList::iterator iter;
  for (iter = available_plugins.begin(); iter != available_plugins.end(); iter++) {
    vw_out() << "  -->  " << (*iter)->getPluginName() << std::endl;
    size_t num_models = (*iter)->getNumModels();
    vw_out() << "    - Num models = " << num_models << std::endl;
    for (size_t i = 0; i < num_models; i++) {
      vw_out() << "      -> " << (*iter)->getModelName(i)
                << ", family =  " << (*iter)->getModelFamily(i) << std::endl;
    }
  }
}
      
// This function is not kept out of the header file to hide CSM dependencies.
/// Look through all of the loaded plugins and find one that is compatible with
///  the provided ISD.
const csm::Plugin* find_plugin_for_isd(csm::Isd const& support_data,
                                       std::string   & model_name,
                                       std::string   & model_family,
                                       bool            show_warnings) {

  // Loop through the available plugins.
  csm::PluginList::iterator iter;
  csm::PluginList plugins = csm::Plugin::getList();
  for (iter = plugins.begin(); iter != plugins.end(); iter++) {
    const csm::Plugin* csm_plugin = (*iter);

    // For each plugin, loop through the available models.
    size_t num_models = csm_plugin->getNumModels();
    for (size_t i = 0; i < num_models; i++) {

      std::string this_model_name = (*iter)->getModelName(i);

      // Check if we can construct a camera with the ISD and this plugin/model.
      csm::WarningList warnings;
      csm::WarningList::const_iterator w_iter;
      if (csm_plugin->canModelBeConstructedFromISD(support_data, this_model_name, &warnings)) {
        model_name   = this_model_name;
        model_family = csm_plugin->getModelFamily(i);
        return csm_plugin; // Found a plugin that will work!
      }
      // Optionally print the reasons why we could not load it.
      if (show_warnings)
        for (w_iter = warnings.begin(); w_iter!=warnings.end(); ++w_iter) {
          vw_out() << "CSM warning: " << w_iter->getMessage() << std::endl;
        }
    } // End loop through models
  } // End loop through plugins

  // Did not find a match!
  model_name   = "";
  model_family = "";
  return 0;
} // End function find_plugin_for_isd


void CsmModel::initialize_plugins() {

  // Only let one thread at a time in here.
  vw::Mutex::Lock lock(csm_init_mutex);

  // If we already have plugins loaded, don't do initialization again.
  csm::PluginList plugins = csm::Plugin::getList();
  if (!plugins.empty())
    return;
  
  //vw_out() << "Initializing CSM plugins...\n";

  // Find all of the available CSM plugin DLL files.
  std::vector<std::string> plugin_files;
  size_t num_plugin_files = find_csm_plugins(plugin_files);
  //vw_out() << "Found " << num_plugin_files << " CSM plugin files.\n";

  // Load all of the plugins.
  for (size_t i = 0; i < num_plugin_files; i++) {
    // Get the DLL in memory, causing it to automatically register itself
    //  with the main Plugin interface.
    vw_out() << "Loading CSM plugin: " << plugin_files[i] << std::endl;
    boost::dll::shared_library lib_usgs(plugin_files[i]);
  }

  //csm::Plugin::setDataDirectory(plugin_folder); // Don't think we need this.

  print_available_models();
}

// Read the semi-major and semi-minor axes
void CsmModel::read_ellipsoid_from_isd(std::string const& isd_path) {

  // Load and parse the json file
  std::ifstream ifs(isd_path);
  json json_isd;
  try {
    ifs >> json_isd;
  } catch(...) {
    vw::vw_throw(vw::ArgumentErr() << "Cannot open file: " << isd_path << "\n");
  }
  
  // Read the semi-major axis
  m_semi_major_axis = 0.0;
  try {
    m_semi_major_axis = json_isd.at("radii").at("semimajor");
  } catch (...){
  }

  // Read the semi-minor axis
  m_semi_minor_axis = 0.0;
  try {
    m_semi_minor_axis = json_isd.at("radii").at("semiminor");
  } catch (...){
  }

  // Read the unit
  std::string unit;
  try {
    unit = json_isd.at("radii").at("unit");
  } catch (...){
  }
  boost::to_lower(unit);

  // Convert from km to m if need be
  if (unit == "km") {
    m_semi_major_axis *= 1000.0;
    m_semi_minor_axis *= 1000.0;
  } else if (unit != "m") {
    vw::vw_throw(vw::ArgumentErr() << "Unknown unit for the ellipsoid radii in "
                  << isd_path << ". The read value is: " << unit);
  }

  // Sanity check
  if (m_semi_major_axis <= 0.0 || m_semi_minor_axis <= 0.0) 
    vw::vw_throw(vw::ArgumentErr() << "Could not read positive semi-major "
                 << "and semi-minor axies from:  " << isd_path
                 << ". The read values are: "
                 << m_semi_major_axis << ' ' << m_semi_minor_axis);
}

/// Load the camera model from an ISD file or model state.
void CsmModel::load_model(std::string const& isd_path) {

  std::string line;
  {
    // Peek inside the file to see if it is an isd or a model state.
    // A model state file starts with an easily identifiable string.
    std::ifstream ifs(isd_path);
    ifs >> line;
  }
  bool is_model_state = (line == UsgsAstroFrameSensorModel::_SENSOR_MODEL_NAME     || 
                         line == UsgsAstroLsSensorModel::_SENSOR_MODEL_NAME        ||
                         line == UsgsAstroPushFrameSensorModel::_SENSOR_MODEL_NAME ||
                         line == UsgsAstroSarSensorModel::_SENSOR_MODEL_NAME);

  if (!is_model_state) 
    CsmModel::load_model_from_isd(isd_path);
  else
    CsmModel::loadModelFromStateFile(isd_path);

  // Read the sun position. Will work for USGSCSM models, but maybe
  // not for others. It is assumed here that the sun does not move
  // noticeably in the sky during the brief time the picture is taken.
  // TODO(oalexan1): Study how important is to compute sun position
  // at every single time. Likely given that a camera shot takes a
  // 1-3 seconds, the Sun can't move that much. 
  std::string modelState = m_gm_model->getModelState();
  nlohmann::json j = stateAsJson(modelState);
  if (j.find("m_sunPosition") != j.end()) {
    std::vector<double> sun_pos = j["m_sunPosition"].get<std::vector<double>>();
    if (sun_pos.size() < 3)
      vw::vw_throw(vw::ArgumentErr() << "The Sun position must be a vector of size >= 3.\n");
    for (size_t it = 0; it < 3; it++) 
      m_sun_position[it] = sun_pos[it];
  }
  
}
  
void CsmModel::load_model_from_isd(std::string const& isd_path) {

  // This only happens the first time it is called.
  initialize_plugins();

  // Load ISD data
  csm::Isd support_data(isd_path);

  CsmModel::read_ellipsoid_from_isd(isd_path);

  // Check each available CSM plugin until we find one that can handle the ISD.
  std::string model_name, model_family;
  const csm::Plugin* csm_plugin = find_plugin_for_isd(support_data, model_name,
                                                      model_family, false);

  // If we did not find a plugin that would work, go through them again and print error
  //  messages for each plugin that fails.
  if (csm_plugin == 0) {
    find_plugin_for_isd(support_data, model_name, model_family, true);
    vw::vw_throw(vw::ArgumentErr() << "Unable to construct a camera model for the ISD file "
                        << isd_path << " using any of the loaded CSM plugins!");
  }

  vw_out() << "Using plugin: " << csm_plugin->getPluginName() 
           << " with model name " << model_name << std::endl;

  // Now try to construct the camera model
  csm::WarningList warnings;
  csm::Model* csm_model
    = csm_plugin->constructModelFromISD(support_data, model_name, &warnings);

  // Error checking
  csm::WarningList::const_iterator w_iter;
  for (w_iter = warnings.begin(); w_iter!=warnings.end(); ++w_iter) {
    vw_out() << "CSM warning: " << w_iter->getMessage() << std::endl;
  }

  // Handle load failure
  if (!csm_model)
    vw::vw_throw(vw::ArgumentErr() << "Failed to load CSM sensor model from file: "
                 << isd_path);
  
  // TODO: Are all sensor models going to be this type (RasterGM)?
  //       Otherwise we can use the result of getModelFamily() to choose the class.
  // Cast the model we got to the child class with the needed functionality.
  csm::RasterGM* gm_model = dynamic_cast<csm::RasterGM*>(csm_model);

   // Handle load failure
  if (!gm_model)
    vw::vw_throw(vw::ArgumentErr() << "Failed to cast CSM sensor model to raster type!");
  
  m_gm_model.reset(gm_model); // The smart pointer will handle memory management
}

/// Load the camera model from a model state written to disk.
/// A model state is obtained from an ISD model by pre-processing
/// and combining its data in a form ready to be used.
void CsmModel::loadModelFromStateFile(std::string const& state_file) {

  // Read the state as one string
  std::ifstream ifs(state_file.c_str());
  std::string model_state;
  ifs.seekg(0, std::ios::end);   
  model_state.reserve(ifs.tellg());
  ifs.seekg(0, std::ios::beg);
  model_state.assign((std::istreambuf_iterator<char>(ifs)),
             std::istreambuf_iterator<char>());
  ifs.close();

  bool recreate_model = true;
  CsmModel::setModelFromStateString(model_state, recreate_model);
}

template<class ModelT>
void setModelFromStateStringAux(bool recreate_model,
                                std::string const& model_state,
                                boost::shared_ptr<csm::RasterGM> & gm_model) {

  if (recreate_model) {

    csm::RasterGM* new_gm_model = NULL;
    ModelT * specific_model = new ModelT;
    specific_model->replaceModelState(model_state);
    new_gm_model = dynamic_cast<csm::RasterGM*>(specific_model);
    
    // Handle load failure
    if (!new_gm_model)
      vw::vw_throw(vw::ArgumentErr() << "Failed to cast linescan model to raster type.");
    
    // This will wipe any preexisting model. Prior gm_model pointer will become invalid.
    gm_model.reset(new_gm_model); 
    
  } else {
    
    // Update existing model. This does not destroy gm_model.
    ModelT * specific_model = static_cast<ModelT*>(gm_model.get());
    if (specific_model == NULL)
      vw::vw_throw(vw::ArgumentErr() << "Incorrect model type passed in.\n");
    specific_model->replaceModelState(model_state);
    
  }

  return;
}
  
/// Load the camera model from a model state written to disk.
/// A model state is obtained from an ISD model by pre-processing
/// and combining its data in a form ready to be used.
/// Use recreate_model = false if desired to just update an existing model.
void CsmModel::setModelFromStateString(std::string const& model_state, bool recreate_model) {
  
  // TODO(oalexan1): Use the usgscsm function
  // constructModelFromState() after that package pushes a new version
  // (currently there are compile-time issues with it).
  
  // See which model to load, then cast it to RasterGM. This could
  // have been simpler if the USGSCSM models shared a base class where
  // all shared functionality would be shared.
  if (model_state.rfind(UsgsAstroFrameSensorModel::_SENSOR_MODEL_NAME, 0) == 0) {

    setModelFromStateStringAux<UsgsAstroFrameSensorModel>
      (recreate_model, model_state, m_gm_model);
    
  } else if (model_state.rfind(UsgsAstroLsSensorModel::_SENSOR_MODEL_NAME, 0) == 0) {
    
    setModelFromStateStringAux<UsgsAstroLsSensorModel>
      (recreate_model, model_state, m_gm_model);
    
  } else if (model_state.rfind(UsgsAstroPushFrameSensorModel::_SENSOR_MODEL_NAME, 0) == 0) {
    
    setModelFromStateStringAux<UsgsAstroPushFrameSensorModel>
      (recreate_model, model_state, m_gm_model);
    
  } else if (model_state.rfind(UsgsAstroSarSensorModel::_SENSOR_MODEL_NAME, 0) == 0) {
    
    setModelFromStateStringAux<UsgsAstroSarSensorModel>
      (recreate_model, model_state, m_gm_model);
    
  } else {
    vw::vw_throw(vw::ArgumentErr() << "Could not create CSM model from state string.\n");
  }

  // Set the semi-axes from json (cannot pull it from the usgs models
  // as these figure as private in some of them).
  auto j = stateAsJson(model_state);
  m_semi_major_axis = j["m_majorAxis"];
  m_semi_minor_axis = j["m_minorAxis"];

  // Sanity check
  if (m_semi_major_axis <= 0.0 || m_semi_minor_axis <= 0.0) 
    vw::vw_throw(vw::ArgumentErr() << "Could not read positive semi-major "
                 << "and semi-minor axies from state string.");
}
  
void CsmModel::throw_if_not_init() const {
  if (!m_gm_model)
    vw_throw(ArgumentErr() << "CsmModel: Sensor model has not been loaded yet!");
}

// TODO: Check all of the warnings

vw::Vector2 CsmModel::get_image_size() const {
  throw_if_not_init();

  csm::ImageVector size = m_gm_model->getImageSize();
  return Vector2(size.samp, size.line);
}

vw::Vector3 CsmModel::target_radii() const {
  return vw::Vector3(m_semi_major_axis,  // x
                     m_semi_major_axis,  // y
                     m_semi_minor_axis); // z
}

Vector2 CsmModel::point_to_pixel(Vector3 const& point) const {
  throw_if_not_init();

  csm::EcefCoord  ecef = vectorToEcefCoord(point);

  double achievedPrecision = -1.0;
  csm::WarningList warnings;
  csm::WarningList * warnings_ptr = NULL;

  // Do not show warnings, it becomes too verbose
  bool show_warnings = false;
  if (show_warnings) 
    warnings_ptr = &warnings;

  csm::ImageCoord imagePt = m_gm_model->groundToImage(ecef, m_desired_precision,
						       &achievedPrecision, warnings_ptr);

  if (show_warnings) {
    csm::WarningList::const_iterator w_iter;
    for (w_iter = warnings.begin(); w_iter!=warnings.end(); ++w_iter) {
      vw_out() << "CSM warning: " << w_iter->getMessage() << std::endl;
    }
  }

  return imageCoordToVector(imagePt) - ASP_TO_CSM_SHIFT;
}

Vector3 CsmModel::pixel_to_vector(Vector2 const& pix) const {
  throw_if_not_init();

  csm::ImageCoord imagePt = vectorToImageCoord(pix + ASP_TO_CSM_SHIFT);

  // Camera center
  csm::EcefCoord  ctr = m_gm_model->getSensorPosition(imagePt);

  // Ground point. Note how we use the 0 height above datum.
  // The precise height value matters only for the SAR model, when the rays
  // are curved, which violates a fundamental assumption in ASP.
  double achievedPrecision = -1.0; // will be modified in the function
  double groundHeight      = 0.0;
  csm::EcefCoord groundPt
    = m_gm_model->imageToGround(imagePt, groundHeight, m_desired_precision,
                                 &achievedPrecision);

  // Normalized direction
  Vector3 dir0 = ecefCoordToVector(groundPt) - ecefCoordToVector(ctr);
  dir0 = dir0 / norm_2(dir0);
  
  return dir0;

#if 0
  // Do not use this since the imageToRemoteImagingLocus() in CSM is
  // buggy as of now and it is not guaranteed long-term to agree with
  // imageToGround().
  
  // This function generates the vector from the camera at the camera origin,
  // there is a different call that gets the vector near the ground.
  // This does not give the right result due to a bug in UsgsAstroSarSensorModel
  csm::EcefLocus locus = m_gm_model->imageToRemoteImagingLocus(imagePt,
                                                                m_desired_precision,
                                                                &achievedPrecision);
  Vector3 dir = ecefVectorToVector(locus.direction);
  
  return dir;
#endif
}

Vector3 CsmModel::camera_center(Vector2 const& pix) const {
  throw_if_not_init();

  csm::ImageCoord imagePt = vectorToImageCoord(pix + ASP_TO_CSM_SHIFT);
  csm::EcefCoord  ecef    = m_gm_model->getSensorPosition(imagePt);

  return ecefCoordToVector(ecef);
}

// Apply a transform to the model state in json format
template<class ModelT>
void applyTransformToState(ModelT const * model,
                           vw::Matrix4x4 const& transform,
                           // Output
                           std::string & modelState) {

  // Applying a scale is not supported in any usgscsm sensors for now.
  double scale = pow(det(transform), 1.0/3.0);
  if (std::abs(scale - 1.0) > 1e-6)
    vw_throw(ArgumentErr()
             << "CSM camera models do not support applying a transform with a scale.\n");

  // Extract the rotation and convert it to ale::Rotation
  vw::Matrix3x3 rotation_matrix = submatrix(transform, 0, 0, 3, 3);
  std::vector<double> rotation_vec;
  for (int row = 0; row < 3; row++) {
    for (int col = 0; col < 3; col++) {
      rotation_vec.push_back(rotation_matrix(row, col));
    }
  }
  ale::Rotation r(rotation_vec);
  
  // Extract the translation
  ale::Vec3d t(transform(0, 3), transform(1, 3), transform(2, 3));
  
  model->applyTransformToState(r, t, modelState);

  return;
}

void applyTransformToState(csm::RasterGM const* gm_model,
                           vw::Matrix4x4 const& transform,
                           // Output
                           std::string & modelState) {
  
  // Need to consider each model type separately
  bool success = false;
  UsgsAstroFrameSensorModel const* frame_model
    = dynamic_cast<UsgsAstroFrameSensorModel const*>(gm_model);
  if (!success && frame_model != NULL) {
    applyTransformToState(frame_model, transform, modelState);
    success = true;
  }
  
  UsgsAstroLsSensorModel const* ls_model
    = dynamic_cast<UsgsAstroLsSensorModel const*>(gm_model);
  if (!success && ls_model != NULL) {
    applyTransformToState(ls_model, transform, modelState);
    success = true;
  }

  UsgsAstroPushFrameSensorModel const* pf_model
    = dynamic_cast<UsgsAstroPushFrameSensorModel const*>(gm_model);
  if (!success && pf_model != NULL) {
    applyTransformToState(pf_model, transform, modelState);
    success = true;
  }

  UsgsAstroSarSensorModel const* sar_model
    = dynamic_cast<UsgsAstroSarSensorModel const*>(gm_model);
  if (!success && sar_model != NULL) {
    applyTransformToState(sar_model, transform, modelState);
    success = true;
  }

  if (!success)
    vw_throw(vw::ArgumentErr()
             << "CsmModel::applyTransformedState(): Unknown CSM model type.\n");
}

// Save model state
void CsmModel::saveState(std::string const& json_state_file) const {
  
  csm::RasterGM const* gm_model
    = dynamic_cast<csm::RasterGM const*>(this->m_gm_model.get());

  std::string modelState = gm_model->getModelState();

  vw_out() << "Writing model state: " << json_state_file << std::endl;
  std::ofstream ofs(json_state_file.c_str());
  ofs << modelState << std::endl;
  ofs.close();

  return;
}

// Apply a transform to the model and save the transformed state as a JSON file.
void CsmModel::saveTransformedState(std::string const& json_state_file,
                                    vw::Matrix4x4 const& transform) const {
  
  csm::RasterGM const* gm_model
    = dynamic_cast<csm::RasterGM const*>(this->m_gm_model.get());

  std::string modelState = gm_model->getModelState();

  applyTransformToState(gm_model, transform,  
                        // Output
                        modelState);
  
  vw_out() << "Writing adjusted model state: " << json_state_file << std::endl;
  std::ofstream ofs(json_state_file.c_str());
  ofs << modelState << std::endl;
  ofs.close();

  return;
}

// Apply a transform to a CSM model
void CsmModel::applyTransform(vw::Matrix4x4 const& transform) {

  csm::RasterGM const* gm_model
    = dynamic_cast<csm::RasterGM const*>(this->m_gm_model.get());
  
  std::string modelState = gm_model->getModelState();
  
  applyTransformToState(gm_model, transform,  
                        // Output
                        modelState);

  bool recreate_model = false; // don't want to destroy the model
  setModelFromStateString(modelState, recreate_model);
}
  
 // Create a CSM frame camera model. Assumes that focal length and optical
 // center are in pixels, the pixel pitch is 1, and no distortion.
 // This requires a lot of bookkeeping. Use cam_test to compare
 // such model with ASP's Pinhole model with same data.
 // That is created as: 
 // vw::camera::PinholeModel pin(C, R, focal_length, focal_length, cx, cy);
 void CsmModel::createFrameModel(int cols, int rows,  // in pixels
        double cx, double cy, // col and row optical center, in pixels
        double focal_length,  // in pixels
        double semi_major_axis, double semi_minor_axis, // in meters
        vw::Vector3 C, // camera center
        vw::Matrix3x3 R) { // camera to world rotation matrix

  // Make a copy of R as an Eigen matrix, and convert to quaternion
  Eigen::Matrix3d R_copy;
  for (int r = 0; r < 3; r++){
    for (int c = 0; c < 3; c++)
      R_copy(r, c) = R(r, c);
  }
  Eigen::Quaterniond q(R_copy);

  // Creating a frame model requires populating a json file
  UsgsAstroFrameSensorModel cam;
  cam.reset();
  std::string state = cam.getModelState();
  nlohmann::json j = stateAsJson(state);

  j["m_sensorName"] = "csm";
  j["m_platformName"] = "csm";
  j["m_majorAxis"] = semi_major_axis;
  j["m_minorAxis"] = semi_minor_axis;
  j["m_minElevation"] = -10000.0; // -10 km
  j["m_maxElevation"] = 10000.0;  // 10 km

  // Use negative signs below and for focal length due to idiosyncrasies of the
  // USGS frame sensor model. These are related to pitch, and we assume
  // pitch = 1.0.
  j["m_iTransL"] = std::vector<double>({0.0, 0.0, -1.0});
  j["m_iTransS"] = std::vector<double>({0.0, -1.0, 0.0});
  j["m_focalLength"] = -focal_length; 
  
  j["m_ccdCenter"] = std::vector<double>({cy, cx}); // note the order (row, col)
  j["m_pixelPitch"] = 1.0; // pixel pitch is set to 1.0
  j["m_nLines"] = rows;
  j["m_nSamples"] = cols;
  j["m_distortionType"] = 0;

  // Need to apply this offset to make CSM agree with ASP's Pinhole
  j["m_startingDetectorLine"] = -0.5;
  j["m_startingDetectorSample"] = -0.5;
    
  // The quantities below don't seem to matter
  j["m_focalLengthEpsilon"] = 1.0; 
  j["m_transX"] = std::vector<double>({0.0, 0.0, -1.0});
  j["m_transY"] = std::vector<double>({0.0, -1.0, 0.0});

  // Set the translation and quaternion. The quaternion is stored as x, y, z, w.
  j["m_currentParameterValue"] = std::vector<double>({C[0], C[1], C[2], 
                                                     q.x(), q.y(), q.z(), q.w()});

  // Update the state string and create the CSM model
  state = cam.getModelName() + "\n" + j.dump(2);
  bool recreate_model = true;
  setModelFromStateString(state, recreate_model);
}

} // end namespace asp

