#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Functions used for the sat_sim.cc tool that are not general enough to put
// somewhere else.

#include <asp/Core/SatSimBase.h>
#include <asp/Core/CameraTransforms.h>
#include <asp/Core/Common.h>
#include <asp/Camera/SatSim.h>
#include <asp/Camera/CsmModel.h>

#include <vw/Core/Stopwatch.h>
#include <vw/Cartography/CameraBBox.h>
#include <vw/Geometry/baseUtils.h>
#include <vw/Cartography/CameraBBox.h>
#include <vw/Cartography/GeoTransform.h>
#include <vw/Camera/PinholeModel.h>

using namespace vw::cartography;
using namespace vw::math;
using namespace vw::geometry;

namespace fs = boost::filesystem;

namespace asp {

// Find a handful of valid DEM values and average them. It helps later when
// intersecting with the DEM, especially for Mars, where the DEM heights ca be
// very far from the datum. 
double findDemHeightGuess(vw::ImageViewRef<vw::PixelMask<float>> const& dem) {

  double height_guess = 0.0;
  bool found = false;
  double sum = 0.0, num = 0.0;
  for (double row = 0; row < dem.rows(); row += dem.rows()/10.0) {
    for (double col = 0; col < dem.cols(); col += dem.cols()/10.0) {
      if (is_valid(dem(col, row))) {
        sum += dem(col, row).child();
        num++;
        if (num > 20) {
          // Those are enough, as going on for too long may take too much time
          found = true;
          break;
        }
      }
    }
    if (found) break;
  }
  if (num > 0) 
    height_guess = sum/num;
    
  return height_guess;

} // End function findDemHeightGuess()

// A function that will read a geo-referenced image, its nodata value,
// and the georeference, and will return a PixelMasked image, the nodata
// value, and the georeference.
// TODO(oalexan1): May need to move this to a more general place.
void readGeorefImage(std::string const& image_file, 
  float & nodata_val, vw::cartography::GeoReference & georef,
  vw::ImageViewRef<vw::PixelMask<float>> & masked_image) {

  // Initial value, in case the image has no nodata field
  nodata_val = std::numeric_limits<float>::quiet_NaN();
  if (!vw::read_nodata_val(image_file, nodata_val))
        vw::vw_out() << "Warning: Could not read the nodata value for: "
                      << image_file << "\nUsing: " << nodata_val << ".\n";

    // Read the image
    vw::vw_out() << "Reading: " << image_file << std::endl;
    vw::DiskImageView<float> image(image_file);
    // Create the masked image
    masked_image = vw::create_mask(image, nodata_val);

    // Read the georeference, and throw an exception if it is missing
    bool has_georef = vw::cartography::read_georeference(georef, image_file);
    if (!has_georef)
      vw::vw_throw(vw::ArgumentErr() << "Missing georeference in: "
                                     << image_file << ".\n");
}

// Compute point on trajectory and along and across track normalized vectors in
// ECEF coordinates, given the first and last proj points and a value t giving
// the position along this line. Produced along and across vectors are
// normalized and perpendicular to each other.
void calcEcefTrajPtAlongAcross(vw::Vector3 const& first_proj,
                               vw::Vector3 const& last_proj,
                               vw::cartography::GeoReference const& dem_georef,
                               double t,
                               double delta,
                               vw::Vector3 const& proj_along,
                               vw::Vector3 const& proj_across,
                               // Outputs
                               vw::Vector3 & P,
                               vw::Vector3 & along,
                               vw::Vector3 & across) {

  // Compute the point on the trajectory, in projected coordinates
  vw::Vector3 proj_pt = first_proj * (1.0 - t) + last_proj * t;

  asp::calcEcefAlongAcross(dem_georef, delta, proj_along, proj_across, proj_pt,
                          // Outputs, as vectors in ECEF
                          along, across);

  // Convert the point along trajectory to ECEF
  P  = vw::cartography::projToEcef(dem_georef, proj_pt);
}

// This is used to signify when the algorithm below fails to find a solution
const double g_big_val = 1e+100;

// Given an orbit given by the first and last camera center positions in
// projected coordinates, a real number t describing the position along this
// line, roll, pitch, and yaw for the camera (relative to nadir), find the z
// direction for the camera (camera look), intersect it with the ground, find
// the DEM pixel location, and return the distance from this location to a given
// pixel location.
double demPixelErr(SatSimOptions const& opt,
                   vw::cartography::GeoReference const& dem_georef,
                   vw::ImageViewRef<vw::PixelMask<float>> dem,
                   vw::Vector3 const& first_proj,
                   vw::Vector3 const& last_proj,
                   vw::Vector3 const& proj_along,
                   vw::Vector3 const& proj_across,
                   double t,
                   double delta, // a small number to move along track
                   double roll, double pitch, double yaw,
                   vw::Vector2 const& pixel_loc,
                   double height_guess,
                   vw::Vector3 & xyz_guess) { // xyz_guess may change

    // Calc position along the trajectory and normalized along and across vectors
    // in ECEF
    vw::Vector3 P, along, across;
    calcEcefTrajPtAlongAcross(first_proj, last_proj, dem_georef, t, delta,
                              proj_along, proj_across, 
                              // Outputs, perpendicular and normal vectors
                              P, along, across);

    // Find the z vector as perpendicular to both along and across
    vw::Vector3 down = vw::math::cross_prod(along, across);
    down = down / norm_2(down);

    // The camera to world rotation
    vw::Matrix3x3 cam2world;
    asp::assembleCam2WorldMatrix(along, across, down, cam2world);
    // Apply the roll-pitch-yaw rotation
    vw::Matrix3x3 R = asp::rollPitchYaw(roll, pitch, yaw);
    cam2world = cam2world * R * asp::rotationXY();

    // Ray from camera to ground going through image center
    vw::Vector3 cam_dir = cam2world * vw::Vector3(0, 0, 1);

    // Find the intersection of this ray with the ground
    bool treat_nodata_as_zero = false;
    bool has_intersection = false;
    double max_abs_tol = std::min(opt.dem_height_error_tol, 1e-14);
    double max_rel_tol = max_abs_tol;
    int num_max_iter = 100;
    vw::Vector3 xyz = vw::cartography::camera_pixel_to_dem_xyz
      (P, cam_dir, dem,
        dem_georef, treat_nodata_as_zero,
        has_intersection, 
        // Below we use a prudent approach. Try to make the solver work
        // hard. It is not clear if this is needed.
        std::min(opt.dem_height_error_tol, 1e-8),
        max_abs_tol, max_rel_tol, 
        num_max_iter, xyz_guess, height_guess);

    if (!has_intersection)
         return g_big_val;

    // Convert to llh
    vw::Vector3 llh = dem_georef.datum().cartesian_to_geodetic(xyz);

    // Find pixel location 
    vw::Vector2 pixel_loc2 = dem_georef.lonlat_to_pixel
      (subvector(llh, 0, 2));

    // If the pixel is outside the DEM, return a big value
    if (!vw::bounding_box(dem).contains(pixel_loc2))
      return g_big_val;

    // At this stage it is safe to update the guess, as we got a good result
    xyz_guess = xyz;

    return norm_2(pixel_loc - pixel_loc2);
}

// A model with the error given by demPixelErr(). The variable will be t,
// which will give the position along the trajectory.
class RayDemPixelLMA : public vw::math::LeastSquaresModelBase<RayDemPixelLMA> {

  SatSimOptions const& m_opt;
  vw::cartography::GeoReference const& m_dem_georef;
  vw::ImageViewRef<vw::PixelMask<float>> m_dem;
  double m_height_guess;
  vw::Vector3 m_first_proj;
  vw::Vector3 m_last_proj;
  vw::Vector3 m_proj_along;
  vw::Vector3 m_proj_across;
  double m_delta, m_param_scale_factor;
  double m_roll, m_pitch, m_yaw;
  vw::Vector2 m_pixel_loc;
  mutable vw::Vector3 m_xyz_guess; // used to speed up the solver, not thread-safe

public:
  typedef vw::Vector<double, 1> result_type;
  typedef vw::Vector<double, 1> domain_type;
  typedef vw::Matrix<double>    jacobian_type; ///< Jacobian form. Auto.

  /// Constructor
  RayDemPixelLMA(SatSimOptions const& opt,
                 vw::cartography::GeoReference const& dem_georef,
                 vw::ImageViewRef<vw::PixelMask<float>> dem,
                 double height_guess,
                 vw::Vector3 const& first_proj,
                 vw::Vector3 const& last_proj,
                 vw::Vector3 const& proj_along,
                 vw::Vector3 const& proj_across,
                 double delta, // a small number to move along track
                 double param_scale_factor, // to go from optimizer units to t in [0, 1]
                 double roll, double pitch, double yaw,
                 vw::Vector2 const& pixel_loc):
    m_opt(opt), m_dem_georef(dem_georef), m_dem(dem), m_height_guess(height_guess),
    m_first_proj(first_proj), m_last_proj(last_proj),
    m_proj_along(proj_along), m_proj_across(proj_across),
    m_delta(delta), m_param_scale_factor(param_scale_factor),
    m_roll(roll), m_pitch(pitch), m_yaw(yaw),
    m_pixel_loc(pixel_loc), m_xyz_guess(vw::Vector3(0, 0, 0)) {}

  // Evaluator operator. The goal is described earlier.
  inline result_type operator()(domain_type const& len) const {

    // See note where param_scale_factor is defined.
    double t = len[0] * m_param_scale_factor;
    double err = demPixelErr(m_opt, m_dem_georef, m_dem, 
                             m_first_proj, m_last_proj,
                             m_proj_along, m_proj_across,
                             t, m_delta, m_roll, m_pitch, m_yaw, m_pixel_loc,
                             m_height_guess,
                             m_xyz_guess); // will change

    result_type result;
    result[0] = err;
    //std::cout << "t = " << t << ", err = " << err << std::endl;
    return result;
  }
};

// Find the location of camera center along the trajectory, in projected
// coordinates, so that the ray from the camera center to the ground goes
// closest to given ground point.
void findBestProjCamLocation
  (SatSimOptions const& opt,
   vw::cartography::GeoReference const& dem_georef,
   vw::ImageViewRef<vw::PixelMask<float>> dem,
   double height_guess, 
   vw::Vector3 const& first_proj, vw::Vector3 const& last_proj,
   vw::Vector3 const& proj_along, vw::Vector3 const& proj_across,
   double delta, double roll, double pitch, double yaw,
   vw::Vector2 const& pixel_loc,
   // Outputs
   vw::Vector3 & best_proj) {

  // Note(oalexan1): This algorithm had issues with convergence. Let eps = 1e-7.
  // This is used in LevenbergMarquardt.h for numerical differentiation. Need to
  // ensure model(len) and model(len + eps) are sufficiently different. For
  // that, ensure that len and len + eps correspond to points in orbit separated
  // by about 1 meter. That is why, we start with t in [0, 1], which
  // parametrizes the orbital segment between first_proj and last_proj, and
  // parametrize using value len, with t = len * param_scale_factor. 
  double eps = 1e-7;
  vw::Vector3 P1 = vw::cartography::projToEcef(dem_georef, first_proj); // t = 0
  vw::Vector3 P2 = vw::cartography::projToEcef(dem_georef, last_proj);  // t = 1
  double d = norm_2(P2 - P1);
  if (d < 1.0)
    vw::vw_throw(vw::ArgumentErr() 
      << "Ensure that the input orbit end points are at least 1 m apart.\n");
  double param_scale_factor = 1.0 / (eps * d);
#if 0
  // Verification that param_scale_factor is correct
  {
    double l1 = 0, l2 = eps;
    double t1 = param_scale_factor * l1; 
    double t2 = param_scale_factor * l2;
    P1 = vw::cartography::projToEcef(dem_georef, first_proj * (1.0 - t1) + last_proj * t1);
    P2 = vw::cartography::projToEcef(dem_georef, first_proj * (1.0 - t2) + last_proj * t2);
    std::cout << "Param scale factor is " << param_scale_factor << std::endl;
    std::cout << "Distance must be 1 meter: " << norm_2(P1 - P2) << std::endl;
  }
#endif

  // Find a spacing in t that corresponds to 1 km movement in orbit.
  // We will use this to find a good initial guess.
  // This is very fragile code.
  // TODO(oalexan1): Find a robust way of finding an initial guess. Sometimes this fails.
  double dt = 1e-3;
  double t1 = -dt, t2 = dt;
  P1 = vw::cartography::projToEcef(dem_georef, first_proj * (1.0 - t1) + last_proj * t1);
  P2 = vw::cartography::projToEcef(dem_georef, first_proj * (1.0 - t2) + last_proj * t2);
  double slope = norm_2(P2 - P1) / (2*dt);
  double spacing = 1000.0 / slope;
#if 0
  // Verification that spacing is correct
  std::cout << "Spacing is " << spacing << std::endl;
  {
    double t1 = 0, t2 = spacing;
    P1 = vw::cartography::projToEcef(dem_georef, first_proj * (1.0 - t1) + last_proj * t1);
    P2 = vw::cartography::projToEcef(dem_georef, first_proj * (1.0 - t2) + last_proj * t2);
    std::cout << "Distance must be 100 meters: " << norm_2(P2 - P1) << std::endl;
  }
#endif

  // Set up the LMA problem
  RayDemPixelLMA model(opt, dem_georef, dem, height_guess, first_proj, last_proj,
                       proj_along, proj_across, delta, param_scale_factor,
                       roll, pitch, yaw, pixel_loc);
  int status = -1;
  double max_abs_tol = 1e-14;
  double max_rel_tol = max_abs_tol;
  int num_max_iter = 100;
  vw::Vector<double, 1> observation; 
  observation[0] = 0; // because we want to minimize the error
  vw::Vector<double, 1> len; len[0] = 0; // initial guess 

  // First need to search around for a good initial guess. This is a bug fix.
  // Number of attempts times spacing in m is 1e+8 m, which is 100,000 km. 
  // Enough for any orbit length.
  // std::cout << "Searching for a good initial guess.\n";
  int attempts = int(1e+8);
  double best_val = g_big_val;
  for (int i = 0; i < attempts; i++) {
    
    // Move towards the positive direction then the negative one
    double curr_best_val = best_val;
    for (int j = -1; j <= 1; j += 2) {
      double t = spacing * i * j;
      vw::Vector<double, 1> len2; 
      len2[0] = t / param_scale_factor;
      double val = model(len2)[0];

      //std::cout << "len, val, attempt = " << len2[0] << ' ' << val << ' ' << i << std::endl;

      if (val < best_val) {
        best_val = val;
        len = len2;
      }
    }
    
    if (curr_best_val == best_val && curr_best_val < g_big_val) {
      // We are not improving anymore, so so stop here, as otherwise
      // we may be going too far.
      break;
    }

  } // end doing attempts

  // Run the optimization with the just-found initial guess
  // std::cout << "Running the solver.\n";
  len = vw::math::levenberg_marquardt(model, len, observation, status, 
      max_abs_tol, max_rel_tol, num_max_iter);

  // Note: The status is ignored here. We will just take whatever the solver
  // outputs, as it may not converge within tolerance. 

#if 0
// Turning this off, as the minimum cost function may be far from zero.
// May need to add some other check here.
  if (std::abs(model(len)[0]) > 1.0) {
    std::cout << "Abs of model value is " << std::abs(model(len)[0]) << std::endl;
    vw::vw_throw(vw::ArgumentErr() << "Error: The solver for finding correct ends of "
      << "orbital segment did not converge to a good solution. Check your DEM, " 
      << "roll, pitch, yaw, and ground path endpoints.\n");
  }
#endif

  // Compute the best location given the just-found position on the segment
  double t = len[0] * param_scale_factor;
  best_proj = first_proj * (1.0 - t) + last_proj * t;
}

// A function to compute orbit length in ECEF given its endpoints in projected
// coordinates. 1e+5 points are used to approximate the orbit length. Should be
// enough. This gets slow when using 1e+6 points.
double calcOrbitLength(vw::Vector3 const& first_proj,
                       vw::Vector3 const& last_proj,
                       vw::cartography::GeoReference const& dem_georef) {

  int num = 1.0e+5;

  // Start of each segment
  vw::Vector3 beg = vw::cartography::projToEcef(dem_georef, first_proj);
  // End of each segment
  vw::Vector3 end = beg;
  double orbitLength = 0.0;

  for (int i = 1; i < num; i++) { // note we start at 1

    double t = double(i) / double(num - 1); 
    // Find the projected position of the current point
    vw::Vector3 curr_proj = first_proj + t * (last_proj - first_proj);
    // Find the ECEF position of the current point
    end = vw::cartography::projToEcef(dem_georef, curr_proj);

    // Add the length of the segment
    orbitLength += norm_2(end - beg);
    // Move to the next segment
    beg = end;
  }

  return orbitLength;
}

// The camera will be constrained by the ground, but not by the roll/pitch/yaw,
// then the orientation will change along the trajectory. Then adjust the along
// and across directions to reflect this. This will adjust the camera direction
// as well.
void cameraAdjustment(vw::Vector2 const& first_ground_pos,
                      vw::Vector2 const& last_ground_pos,
                      double t,
                      vw::cartography::GeoReference const& dem_georef,
                      vw::ImageViewRef<vw::PixelMask<float>> dem,
                      vw::Vector3 const& P, // camera center
                      // Outputs
                      vw::Vector3 & along, vw::Vector3 & across) {

  // Create interpolated DEM with bilinear interpolation with invalid pixel 
  // edge extension
  vw::PixelMask<float> nodata_mask = vw::PixelMask<float>(); // invalid value
  nodata_mask.invalidate();
  auto interp_dem = vw::interpolate(dem, vw::BilinearInterpolation(),
  vw::ValueEdgeExtension<vw::PixelMask<float>>(nodata_mask));

  // The camera will be constrained by the ground, but not by the roll/pitch/yaw,
  // then the orientation will change along the trajectory.
  vw::Vector2 ground_pix = first_ground_pos * (1.0 - t) + last_ground_pos * t;

  // Find the projected position along the ground path
  vw::Vector3 ground_proj_pos;
  subvector(ground_proj_pos, 0, 2) = dem_georef.pixel_to_point(ground_pix); // x and y

  auto val = interp_dem(ground_pix[0], ground_pix[1]);
  if (!is_valid(val))
    vw::vw_throw(vw::ArgumentErr() 
      << "Could not interpolate into the DEM along the ground path.\n");
  ground_proj_pos[2] = val.child(); // z

  // Convert the ground point to ECEF
  vw::Vector3 G = vw::cartography::projToEcef(dem_georef, ground_proj_pos);

  // Find the ground direction
  vw::Vector3 ground_dir = G - P;
  if (norm_2(ground_dir) < 1e-6)
    vw::vw_throw(vw::ArgumentErr()
      << "The ground position is too close to the camera.\n");

  // Normalize      
  along = along / norm_2(along);
  ground_dir = ground_dir / norm_2(ground_dir);

  // Adjust the along-track direction to make it perpendicular to ground dir
  along = along - dot_prod(ground_dir, along) * ground_dir;

  // Find 'across' as y direction, given that 'along' is x, and 'ground_dir' is z
  across = -vw::math::cross_prod(along, ground_dir);

  // Make these vectors have norm 1, and make across perpendicular to along
  // Should already be that way by now, but do it just in case
  asp::normalizeOrthogonalizeAlongAcross(along, across);
}

// Adjust the orbit end point and set the number of cameras given the frame rate
// This is a bit tricky because need to sample finely the orbit
void adjustForFrameRate(SatSimOptions                  const& opt,
                        vw::cartography::GeoReference const& dem_georef,
                        vw::Vector3                   const& first_proj,
                        // Outputs
                        vw::Vector3                        & last_proj, // in/out
                        int                                & num_cameras) {

  // Initialize the outputs, this value will change
  num_cameras = 0;

  // Orbit length in meters. Throw an error if getting an orbit of length 0,
  // as that suggests there was a failure in finding in orbit end points.
  double orbit_len = calcOrbitLength(first_proj, last_proj, dem_georef);
  if (orbit_len <= 0.0)
    vw::vw_throw(vw::ArgumentErr() << "Failure in computing orbit end points.\n");
  double period = opt.velocity / opt.frame_rate;

  // It is important to let the user know this
  vw::vw_out() << std::setprecision(17) 
      << "Distance between successive cameras = velocity / frame_rate = "
      << period << " meters.\n";

  // Number of cameras. Add 1 because we need to include the last camera.
  num_cameras = int(orbit_len / period) + 1;

  // Sanity checks. It is fine to have one single camera, but that is not usual.
  if (num_cameras < 1)
    vw::vw_throw(vw::ArgumentErr() << "The number of cameras must be at least 1.\n");
  if (num_cameras == 1)
    vw::vw_out(vw::WarningMessage) << "Warning: Creating only one camera sample.\n";

  // Update the orbit length
  orbit_len = period * (num_cameras - 1.0);

  // Sanity check, important for the work below. It is fine for first and last 
  // proj to be in the same location, but that is not usual.
  if (norm_2(last_proj - first_proj) < 1e-6)
    vw::vw_out(vw::WarningMessage) << "Warning: The first and last camera positions are too close. Check your inputs.\n";

  // Travel along the orbit in very small increments. Return the last point
  // before exceeding the orbit length. 
  int num = 1000000; // one million samples along the orbit should be enough
  vw::Vector3 beg = vw::cartography::projToEcef(dem_georef, first_proj);
  vw::Vector3 end = beg;
  vw::Vector3 out_proj = first_proj; // will keep the result here
  double curr_len = 0.0;
  int i = 1;
  while (1) {

    // Find the projected position of the current point
    double t = double(i) / double(num - 1); 
    vw::Vector3 curr_proj = first_proj + t * (last_proj - first_proj);

    // Find the ECEF position of the current point and distance from previous
    end = vw::cartography::projToEcef(dem_georef, curr_proj);
    double curr_dist = norm_2(end - beg);

    if (curr_len + curr_dist > orbit_len)
      break; // done, exceeded orbit length, will keep the previous point in out_proj
    
    curr_len += curr_dist;
    beg = end;
    i++;
    out_proj = curr_proj;

    // Sanity check
    if (i >= 100 * num) {
      vw::vw_out() << "Warning: Could not find the last camera along the orbit. Perhaps the frame rate is too low.\n";
      break;
    }

  }

  // Update the last orbit point, in projected coords
  last_proj = out_proj;
}

// Given the direction going from first_proj to last_proj, and knowing
// that we are at curr_proj, find if we are before or after orig_first_proj.
// Return 1 if after, -1 if before, 0 if at that point. 
double findDirectionAlongOrbit(vw::Vector3 const& orig_first_proj,
                                vw::Vector3 const& first_proj,
                                vw::Vector3 const& last_proj,
                                vw::Vector3 const& curr_proj) {
  double sign = 0.0;
  vw::Vector3 dir1 = last_proj - first_proj;
  double len1 = norm_2(dir1);
  if (len1 != 0)
    dir1 = dir1 / len1;

  // Handle the case when we are at orig_first_proj
  vw::Vector3 dir2 = curr_proj - orig_first_proj;
  double len2 = norm_2(dir2);
  if (len2 == 0)
    return 0;

   dir2 = dir2 / len2;
   if (norm_2(dir1 - dir2) <= norm_2(dir1 + dir2))
     sign = 1.0; // curr_proj is after orig_first_proj
   else  
    sign = -1.0; // curr_proj is before orig_first_proj

  return sign;
}

// Calc the jitter amplitude at a given location along the orbit. We will
// accumulate over all frequencies. We measure the orbit length from original
// user-set starting point, even if the first camera is not there. That because
// we will end up using different orbital segments for different roll, pitch,
// and yaw, but we want to always measure from same starting point. Use a
// different amplitude and phase shift for roll, pitch, and yaw. But all these
// share the same set of frequencies.
vw::Vector3 calcJitterAmplitude(SatSimOptions const& opt,
                              vw::Vector3 const& orig_first_proj, // user-set
                              vw::Vector3 const& first_proj, // first actual camera
                              vw::Vector3 const& last_proj,  // last actual camera
                              vw::cartography::GeoReference const& dem_georef,
                              double t) {

  vw::Vector3 amp(0, 0, 0);

  // Current postion in projected coordinates and height above datum for it
  vw::Vector3 curr_proj = first_proj * (1.0 - t) + last_proj * t;
  double height_above_datum = curr_proj[2];

  // Length of the orbit from starting point, orig_first_proj, before
  // adjustment for roll, pitch, and yaw. This way when different orbital
  // segments are used, for different roll, pitch, and yaw, d will not
  // always start as 0 at the beginning of each segment. TODO(oalexan1):
  // Better calculate orbit length from prev proj to curr proj, then add up
  // to previous value.
  double dist = calcOrbitLength(orig_first_proj, curr_proj, dem_georef);

  // Find if we are before or after orig_first_proj. This will multiply 'dist' below.
  double sign = findDirectionAlongOrbit(orig_first_proj, first_proj, last_proj, curr_proj);

  for (size_t freq_iter = 0; freq_iter < opt.jitter_frequency.size(); freq_iter++) {
    double f = opt.jitter_frequency[freq_iter];
    double v = opt.velocity;
    double T = v / f; // period in meters

    // Iterate over roll, pitch, and yaw
    for (int c = 0; c < 3; c++) {
      int index = 3 * freq_iter + c;
      double a = 0.0;
      // We have either horizontal uncertainty or jitter amplitude.
      if (!opt.horizontal_uncertainty.empty()) {
        // jitter amplitude as angular uncertainty given ground uncertainty
        a = atan(opt.horizontal_uncertainty[c] / height_above_datum);
        // Covert to degrees
        a = a * 180.0 / M_PI;
      } else {
        // Amplitude in micro radians
        a = opt.jitter_amplitude[index];
        // Convert to radians
        a = a * 1e-6;
        // Convert to degrees
        a = a * 180.0 / M_PI;
      }

      // Compute the jitter, in degrees. Add the phase.
      amp[c] += a * sin(sign * dist * 2.0 * M_PI / T + opt.jitter_phase[index]);
    }

  } // End loop through frequencies

  return amp;
}

// A function that will take as input the endpoints and will compute the
// satellite trajectory and along track/across track/down directions in ECEF,
// which will give the camera to world rotation matrix.
// The key observation is that the trajectory will be a straight edge in
// projected coordinates so will be computed there first. In some usage
// modes we will adjust the end points of the trajectory along the way.
void calcTrajectory(SatSimOptions & opt,
                    vw::cartography::GeoReference const& dem_georef,
                    vw::ImageViewRef<vw::PixelMask<float>> dem,
                    double height_guess,
                    // Outputs
                    double                       & orbit_len,
                    std::map<int, vw::Vector3>   & trajectory,
                    std::map<int, vw::Matrix3x3> & cam2world,
                    std::map<int, vw::Matrix3x3> & cam2world_no_jitter,
                    std::map<int, vw::Matrix3x3> & ref_cam2world) {

  // Initialize the outputs
  orbit_len = 0.0;
  trajectory.clear();
  cam2world.clear();
  cam2world_no_jitter.clear();
  ref_cam2world.clear();

  // Convert the first and last camera center positions to projected coordinates
  vw::Vector3 first_proj, last_proj;
  subvector(first_proj, 0, 2) = dem_georef.pixel_to_point
      (vw::math::subvector(opt.first, 0, 2)); // x and y
  first_proj[2] = opt.first[2]; // z
  subvector(last_proj, 0, 2) = dem_georef.pixel_to_point
      (vw::math::subvector(opt.last,  0, 2)); // x and y
  last_proj[2] = opt.last[2]; // z

  // Direction along the edge in proj coords (along track direction),
  // and then the across track direction
  vw::Vector3 proj_along, proj_across;
  asp::calcProjAlongAcross(first_proj, last_proj, proj_along, proj_across);
  
  // A small number to help convert directions from being in projected space to
  // ECEF (the transform between these is nonlinear). Do not use a small value,
  // as in ECEF these will be large numbers and we may have precision issues.
  // The value 0.01 was tested well.
  double delta = asp::satSimDelta(); // in meters

  bool have_ground_pos = !std::isnan(norm_2(opt.first_ground_pos)) &&  
      !std::isnan(norm_2(opt.last_ground_pos));
  bool have_roll_pitch_yaw = !std::isnan(opt.roll) && !std::isnan(opt.pitch) &&
      !std::isnan(opt.yaw);

  // Starting point of orbit before we adjust it to match the desired
  // ground locations and roll/pitch/yaw angles.
  vw::Vector3 orig_first_proj = first_proj;

  if (have_ground_pos && have_roll_pitch_yaw) {
    // Find best starting and ending points for the orbit given desired
    // ground locations and roll/pitch/yaw angles.
    // Print a message as this step can take a while
    vw::vw_out() << "Estimating orbital segment endpoints given ground constraints.\n";
    vw::Vector3 first_best_cam_loc_proj;
    vw::Stopwatch sw1;
    sw1.start();
    findBestProjCamLocation(opt, dem_georef, dem, height_guess, first_proj, last_proj,
                            proj_along, proj_across, delta, 
                            opt.roll, opt.pitch, opt.yaw,
                            opt.first_ground_pos, first_best_cam_loc_proj);
    sw1.stop();
    vw::vw_out() << "Elapsed time for starting endpoint: " << sw1.elapsed_seconds() << " s.\n";
    // Same thing for the last camera
    vw::Stopwatch sw2;
    sw2.start();
    vw::Vector3 last_best_cam_loc_proj;
    findBestProjCamLocation(opt, dem_georef, dem, height_guess, first_proj, last_proj,
                            proj_along, proj_across, delta, 
                            opt.roll, opt.pitch, opt.yaw,
                            opt.last_ground_pos, last_best_cam_loc_proj);
    sw2.stop();
    vw::vw_out() << "Elapsed time for ending endpoint: " << sw2.elapsed_seconds() << " s.\n";
    // Overwrite the first and last camera locations in projected coordinates
    // with the best ones
    first_proj = first_best_cam_loc_proj;
    last_proj  = last_best_cam_loc_proj;
  }                  

  if (!std::isnan(opt.frame_rate)) {
    // Adjust the orbit end point and set the number of cameras given the frame rate
    adjustForFrameRate(opt, dem_georef, first_proj, 
                      // outputs
                      last_proj, opt.num_cameras);                      
  }

  orbit_len = calcOrbitLength(first_proj, last_proj, dem_georef); // will be passed out

  // Good to print these
  vw::vw_out() << "Orbit length between first and last adjusted cameras: " 
     << orbit_len << " meters.\n"; 
  vw::vw_out() << "Number of camera samples: " << opt.num_cameras << "." << std::endl;

  // We did a sanity check to ensure that when opt.jitter_frequency is set,
  // opt.velocity and and opt.horizontal_uncertainty are also set and not NaN.
  bool model_jitter = (!std::isnan(opt.jitter_frequency[0]));

  // Find the trajectory, as well as points in the along track and across track 
  // directions in the projected space
  vw::vw_out() << "Computing the camera poses.\n"; 
  std::vector<vw::Vector3> along_track(opt.num_cameras), across_track(opt.num_cameras);

  // For linescan cameras we want to go beyond the positions and orientations
  // needed for the first and last image line, to have room for interpolation
  // and jitter. For Pinhole cameras we do not need this.
  int first_pos = 0, last_pos = opt.num_cameras; // stop before last
  if (opt.sensor_type == "linescan") {
     // Double the number of cameras, half of extra ones going beyond image lines
     first_pos = -opt.num_cameras/2;
     last_pos  = 2 * opt.num_cameras + first_pos;
  }

  // Print progress
  vw::TerminalProgressCallback tpc("asp", "\t--> ");
  double inc_amount = 1.0 / (last_pos - first_pos);
  tpc.report_progress(0);

  for (int i = first_pos; i < last_pos; i++) {

    // Calc position along the trajectory and normalized along and across vectors
    // in ECEF. Produced along and across vectors are normalized and perpendicular
    // to each other.
    double t = double(i) / std::max(double(opt.num_cameras - 1.0), 1.0);
    vw::Vector3 P, along, across;
    calcEcefTrajPtAlongAcross(first_proj, last_proj, dem_georef, t, delta,
                              proj_along, proj_across, 
                              // Outputs, in ECEF
                              P, along, across);

    // Adjust the camera if constrained by the ground but not by roll/pitch/yaw
    if (have_ground_pos && !have_roll_pitch_yaw)
      cameraAdjustment(opt.first_ground_pos, opt.last_ground_pos, t, dem_georef, dem, P, 
                       // outputs, will be normalized and perpendicular to each other
                       along, across);
    
    // Find the z vector as perpendicular to both along and across
    vw::Vector3 down = vw::math::cross_prod(along, across);
    down = down / norm_2(down);

    // Trajectory
    trajectory[i] = P;
    
    // The camera to world rotation has these vectors as the columns
    asp::assembleCam2WorldMatrix(along, across, down, cam2world[i]);

    // Save this before applying adjustments as below. These two 
    // have some important differences, as can be seen below.
    ref_cam2world[i] = cam2world[i];
    cam2world_no_jitter[i] = cam2world[i];

    // See if to apply the jitter
    vw::Vector3 jitter_amp(0, 0, 0);
    if (model_jitter)
      jitter_amp = calcJitterAmplitude(opt, orig_first_proj, first_proj, last_proj, 
                                       dem_georef, t);                       

    // If to apply a roll, pitch, yaw rotation
    if (have_roll_pitch_yaw) {
      vw::Matrix3x3 R = asp::rollPitchYaw(opt.roll  + jitter_amp[0], 
                                          opt.pitch + jitter_amp[1], 
                                          opt.yaw   + jitter_amp[2]);
      cam2world[i] = cam2world[i] * R;
    }

    // The rotation without jitter
    vw::Matrix3x3 R0 = vw::math::identity_matrix<3>();
    if (have_roll_pitch_yaw)
      R0 = asp::rollPitchYaw(opt.roll, opt.pitch, opt.yaw);
    cam2world_no_jitter[i] = cam2world_no_jitter[i] * R0;

    // In either case apply the in-plane rotation from camera to satellite frame
    cam2world[i] = cam2world[i] * asp::rotationXY();
    cam2world_no_jitter[i] = cam2world_no_jitter[i] * asp::rotationXY();

    tpc.report_incremental_progress(inc_amount);
  }
  tpc.report_finished();

  return;
}

// Generate a prefix that will be used for image names and camera names
std::string genPrefix(SatSimOptions const& opt, int i) {
  return opt.out_prefix + "-" + num2str(10000 + i);
}

// Generate a prefix that will be used for reference camera, without 
// roll, pitch, yaw, jitter, or rotation from camera to satellite frame
std::string genRefPrefix(SatSimOptions const& opt, int i) {
  return opt.out_prefix + "-ref-" + num2str(10000 + i);
}

// A function to read Pinhole cameras from disk
void readPinholeCameras(SatSimOptions const& opt, 
    std::vector<std::string> & cam_names,
    std::vector<vw::CamPtr> & cams) {

  // Read the camera names
  vw::vw_out() << "Reading: " << opt.camera_list << std::endl;
  asp::read_list(opt.camera_list, cam_names);

  // Sanity check
  if (cam_names.empty())
    vw::vw_throw(vw::ArgumentErr() << "No cameras were found.\n");

  cams.resize(cam_names.size());
  for (int i = 0; i < int(cam_names.size()); i++)
    cams[i] = vw::CamPtr(new vw::camera::PinholeModel(cam_names[i]));

  return;
}

// Check if we do a range
bool skipCamera(int i, SatSimOptions const& opt) {

  if (opt.first_index >= 0 && opt.last_index >= 0 &&
     (i < opt.first_index || i >= opt.last_index))
       return true;
  return false;
}

// A function to create and save Pinhole cameras. Assume no distortion, and pixel
// pitch = 1.
void genPinholeCameras(SatSimOptions       const & opt,
            vw::cartography::GeoReference  const & dem_georef,
            std::map<int, vw::Vector3>     const & trajectory,
            std::map<int, vw::Matrix3x3>   const & cam2world,
            std::map<int, vw::Matrix3x3>   const & ref_cam2world,
            // outputs
            std::vector<std::string>             & cam_names,
            std::vector<vw::CamPtr>              & cams) {

  // Ensure we have as many camera positions as we have camera orientations
  if (trajectory.size() != cam2world.size() || trajectory.size() != ref_cam2world.size())
    vw::vw_throw(vw::ArgumentErr()
      << "Expecting as many camera positions as camera orientations.\n");

  cams.resize(trajectory.size());
  cam_names.resize(trajectory.size());
  for (int i = 0; i < int(trajectory.size()); i++) {

    // Always create the cameras, but only save them if we are not skipping
    asp::CsmModel * csmPtr = NULL;
    vw::camera::PinholeModel *pinPtr = NULL; 
    vw::cartography::Datum d = dem_georef.datum();
    if (opt.save_as_csm) {
      csmPtr = new asp::CsmModel;
      csmPtr->createFrameModel(opt.image_size[0], opt.image_size[1],
                              opt.optical_center[0], opt.optical_center[1],
                              opt.focal_length, 
                              d.semi_major_axis(), d.semi_minor_axis(),
                              asp::mapVal(trajectory, i), 
                              asp::mapVal(cam2world, i));
      cams[i] = vw::CamPtr(csmPtr); // will own this pointer
    } else {
      pinPtr = new vw::camera::PinholeModel(asp::mapVal(trajectory, i), 
                                           asp::mapVal(cam2world, i),
                                           opt.focal_length, opt.focal_length,
                                           opt.optical_center[0], opt.optical_center[1]);
      cams[i] = vw::CamPtr(pinPtr); // will own this pointer
    }

    // This is useful for understanding things in the satellite frame
    vw::camera::PinholeModel pinRefCam;
    asp::CsmModel csmRefCam;
    if (opt.save_ref_cams) {
      if (opt.save_as_csm) 
        csmRefCam.createFrameModel(opt.image_size[0], opt.image_size[1],
                                   opt.optical_center[0], opt.optical_center[1],
                                   opt.focal_length, 
                                   d.semi_major_axis(), d.semi_minor_axis(),
                                   asp::mapVal(trajectory, i), 
                                   asp::mapVal(ref_cam2world, i));
      else
        pinRefCam = vw::camera::PinholeModel(asp::mapVal(trajectory, i), 
                                             asp::mapVal(ref_cam2world, i),
                                             opt.focal_length, opt.focal_length,
                                             opt.optical_center[0], opt.optical_center[1]);
    }

    std::string ext;
    if (opt.save_as_csm)
      ext = ".json";
    else
      ext = ".tsai"; 

    std::string camName = genPrefix(opt, i) + ext;
    cam_names[i] = camName;

    // Check if we do a range
    if (skipCamera(i, opt)) continue;

    vw::vw_out() << "Writing: " << camName << std::endl;
    if (opt.save_as_csm) 
      csmPtr->saveState(camName);
    else
      pinPtr->write(camName);

    if (opt.save_ref_cams) {
      std::string refCamName = genRefPrefix(opt, i) + ext;
      vw::vw_out() << "Writing: " << refCamName << std::endl;
      if (opt.save_as_csm)
        csmRefCam.saveState(refCamName);
      else
       pinRefCam.write(refCamName);
    }
  }

  // Write the list of cameras only if we are not skipping the first camera
  // Otherwise the same file may be written by multiple processes. 
  if (!skipCamera(0, opt)) {
    std::string cam_list = opt.out_prefix + "-cameras.txt"; 
    vw::vw_out() << "Writing: " << cam_list << std::endl;
    asp::write_list(cam_list, cam_names);
  } else {
     // Print a warning message that the list won't be saved
     vw::vw_out(vw::WarningMessage) << "The camera list is saved only when " 
        << "--first-index is 0, to avoid a race condition.\n";
  }

  return;
}

// Bring crops in memory. It greatly helps with multi-threading speed.  
// This function is used only for small tiles, to avoid running out of memory
// (which did happen).
void setupCroppedDemAndOrtho(vw::Vector2 const& image_size,
    vw::CamPtr const& cam,
    vw::ImageViewRef<vw::PixelMask<float>> const& dem,
    vw::cartography::GeoReference const& dem_georef,
    vw::ImageViewRef<vw::PixelMask<float>> const& ortho,
    vw::cartography::GeoReference const& ortho_georef,
    // Outputs
    vw::ImageView<vw::PixelMask<float>> & crop_dem,
    vw::cartography::GeoReference & crop_dem_georef,
    vw::ImageView<vw::PixelMask<float>> & crop_ortho,
    vw::cartography::GeoReference & crop_ortho_georef) {

    // Find the bounding box of the dem and ortho portions seen in the camera,
    // in projected coordinates
    float mean_gsd = 0.0;    
    bool quick = true; // Assumes a big DEM fully containing the image    
    vw::BBox2 dem_box;
    try {
      // This is a bugfix. The camera_bbox function can fail if we want an
      // extent that does not fit fully with the input DEM/orthoimage.
      // We could use above quick = false, but then we'd get only partial
      // synthetic images, which isn't good.
      dem_box = vw::cartography::camera_bbox(dem, dem_georef, dem_georef,
        cam, image_size[0], image_size[1], mean_gsd, quick);
    } catch (const vw::Exception& e) {
       vw::vw_throw(vw::ArgumentErr() << "sat_sim: Failed to compute a synthetic image. "
       << "The most likely cause is that the desired image is out of bounds given "
       << "the input DEM and orthoimage.\n");
    }

    vw::cartography::GeoTransform d2o(dem_georef, ortho_georef);
    vw::BBox2 ortho_box = d2o.point_to_point_bbox(dem_box);

    // Find the DEM pixel box and expand it in case there was some inaccuracies
    // in finding the box
    vw::BBox2i dem_pixel_box = dem_georef.point_to_pixel_bbox(dem_box);
    int expand = 50;
    dem_pixel_box.expand(expand);
    dem_pixel_box.crop(vw::bounding_box(dem));

    // Same for the ortho
    vw::BBox2i ortho_pixel_box = ortho_georef.point_to_pixel_bbox(ortho_box);
    ortho_pixel_box.expand(expand);
    ortho_pixel_box.crop(vw::bounding_box(ortho));

    // Crop
    crop_dem = vw::crop(dem, dem_pixel_box);
    crop_dem_georef = crop(dem_georef, dem_pixel_box);
    crop_ortho = vw::crop(ortho, ortho_pixel_box);
    crop_ortho_georef = crop(ortho_georef, ortho_pixel_box);
}

// Create a synthetic image with multiple threads
typedef vw::ImageView<vw::PixelMask<float>> ImageT;
class SynImageView: public vw::ImageViewBase<SynImageView> {
  
  typedef typename ImageT::pixel_type PixelT;
  SatSimOptions const& m_opt;
  vw::CamPtr m_cam;
  vw::cartography::GeoReference m_dem_georef; // make a copy to be thread-safe
  vw::ImageViewRef<vw::PixelMask<float>> const& m_dem;
  double m_height_guess;
  vw::cartography::GeoReference m_ortho_georef; // make a copy to be thread-safe
  vw::ImageViewRef<vw::PixelMask<float>> const& m_ortho;
  float m_ortho_nodata_val;

public:
  SynImageView(SatSimOptions const& opt,
               vw::CamPtr    const& cam,
               vw::cartography::GeoReference   const& dem_georef,
               vw::ImageViewRef<vw::PixelMask<float>> dem,
               double height_guess,
               vw::cartography::GeoReference   const& ortho_georef,
               vw::ImageViewRef<vw::PixelMask<float>> ortho,
               float ortho_nodata_val):
                m_opt(opt), m_cam(cam), 
                m_dem_georef(dem_georef), m_dem(dem),
                m_height_guess(height_guess),
                m_ortho_georef(ortho_georef), m_ortho(ortho),
                m_ortho_nodata_val(ortho_nodata_val) {}

  typedef PixelT pixel_type;
  typedef PixelT result_type;
  typedef vw::ProceduralPixelAccessor<SynImageView> pixel_accessor;

  inline vw::int32 cols() const { return m_opt.image_size[0]; }
  inline vw::int32 rows() const { return m_opt.image_size[1]; }
  inline vw::int32 planes() const { return 1; }

  inline pixel_accessor origin() const { return pixel_accessor(*this, 0, 0); }

  inline pixel_type operator()( double/*i*/, double/*j*/, vw::int32/*p*/ = 0 ) const {
    vw::vw_throw(vw::NoImplErr() 
      << "SynImageView::operator()(...) is not implemented");
    return pixel_type();
  }

  typedef vw::CropView<vw::ImageView<pixel_type>> prerasterize_type;
  inline prerasterize_type prerasterize(vw::BBox2i const& bbox) const {

    // Expand the box a bit, to help with interpolation in the ortho image later
    vw::BBox2i extra_bbox = bbox;
    extra_bbox.expand(10);
    extra_bbox.crop(vw::BBox2i(0, 0, cols(), rows()));
    vw::Vector2 crop_start = extra_bbox.min();
    vw::Vector2 crop_image_size = extra_bbox.max() - extra_bbox.min();

    // Must adjust the camera to be able work with the current box, which 
    // does not necessarily start at (0,0). We only adjust the starting
    // position, and not any other params
    vw::Vector3 translation(0, 0, 0);
    vw::Quat    rotation(vw::math::identity_matrix<3>());
    vw::Vector2 pixel_offset = crop_start;
    double      scale = 1.0;
    vw::CamPtr crop_cam(new vw::camera::AdjustedCameraModel(m_cam, translation, rotation, 
                        pixel_offset, scale));

    // Bring crops in memory. It greatly helps with multi-threading speed.  
    vw::cartography::GeoReference crop_dem_georef; // make a copy to be thread-safe
    vw::ImageView<vw::PixelMask<float>> crop_dem;
    vw::cartography::GeoReference crop_ortho_georef; // make a copy to be thread-safe
    vw::ImageView<vw::PixelMask<float>> crop_ortho;
    setupCroppedDemAndOrtho(crop_image_size,
      crop_cam, m_dem, m_dem_georef, m_ortho, m_ortho_georef, 
      // Outputs
      crop_dem, crop_dem_georef, crop_ortho, crop_ortho_georef);

    // Create interpolated image with bicubic interpolation with invalid pixel 
    // edge extension
    vw::PixelMask<float> nodata_mask = vw::PixelMask<float>(); // invalid value
    nodata_mask.invalidate();
    auto interp_ortho = vw::interpolate(crop_ortho, vw::BicubicInterpolation(),
                                        vw::ValueEdgeExtension<vw::PixelMask<float>>(nodata_mask));

    // The location where the ray intersects the ground. We will use each obtained
    // location as initial guess for the next ray. This may not be always a great
    // guess, but it is better than starting nowhere. It should work decently
    // if the camera is high, and with a small footprint on the ground.
    vw::Vector3 xyz_guess(0, 0, 0);

    vw::ImageView<result_type> tile(bbox.width(), bbox.height());

    for (int col = bbox.min().x(); col < bbox.max().x(); col++) {
      for (int row = bbox.min().y(); row < bbox.max().y(); row++) {

        // These will use to index into the tile 
        int c = col - bbox.min().x();
        int r = row - bbox.min().y();

        // Start with an invalid pixel
        tile(c, r) = vw::PixelMask<float>();
        tile(c, r).invalidate();

        // Here use the full image pixel indices
        vw::Vector2 pix(col, row);
        // Also use original camera
        vw::Vector3 cam_ctr = m_cam->camera_center(pix);
        vw::Vector3 cam_dir = m_cam->pixel_to_vector(pix);

        // Intersect the ray going from the given camera pixel with a DEM
        // Use xyz_guess as initial guess and overwrite it with the new value
        bool treat_nodata_as_zero = false;
        bool has_intersection = false;
        double max_abs_tol = std::min(m_opt.dem_height_error_tol, 1e-14);
        double max_rel_tol = max_abs_tol;
        int num_max_iter = 100;
        vw::Vector3 xyz = vw::cartography::camera_pixel_to_dem_xyz
          (cam_ctr, cam_dir, crop_dem,
            crop_dem_georef, treat_nodata_as_zero,
            has_intersection, m_opt.dem_height_error_tol, 
            max_abs_tol, max_rel_tol, 
            num_max_iter, xyz_guess, m_height_guess);

        if (!has_intersection)
          continue; // will result in nodata pixels

        // Update the guess for nex time, now that we have a valid intersection point
        xyz_guess = xyz;

        // Find the texture value at the intersection point by interpolation.
        // This will result in an invalid value if if out of range or if the
        // image itself has invalid pixels.
        vw::Vector3 llh = crop_dem_georef.datum().cartesian_to_geodetic(xyz);
        vw::Vector2 ortho_pix = crop_ortho_georef.lonlat_to_pixel
                                 (vw::Vector2(llh[0], llh[1]));
        tile(c, r) = interp_ortho(ortho_pix[0], ortho_pix[1]);
      }
    }

    return prerasterize_type(tile, -bbox.min().x(), -bbox.min().y(),
                             cols(), rows());
  }

  template <class DestT>
  inline void rasterize(DestT const& dest, vw::BBox2i bbox) const {
    vw::rasterize(prerasterize(bbox), dest, bbox);
  }
};

// Generate images by projecting rays from the sensor to the ground
void genImages(SatSimOptions const& opt,
    bool external_cameras,
    std::vector<std::string> const& cam_names,
    std::vector<vw::CamPtr> const& cams,
    vw::cartography::GeoReference const& dem_georef,
    vw::ImageViewRef<vw::PixelMask<float>> dem,
    double height_guess,
    vw::cartography::GeoReference const& ortho_georef,
    vw::ImageViewRef<vw::PixelMask<float>> ortho,
    float ortho_nodata_val) {

  vw::vw_out() << "Generating images.\n";

  // Generate image names from camera names by replacing the extension
  std::vector<std::string> image_names;
  image_names.resize(cam_names.size());
  for (int i = 0; i < int(cam_names.size()); i++) {
    if (external_cameras)
      image_names[i] = opt.out_prefix + "-" 
      + fs::path(cam_names[i]).filename().replace_extension(".tif").string();
    else if (opt.sensor_type == "pinhole")
      image_names[i] = genPrefix(opt, i) + ".tif";
    else // just replace the extension
     image_names[i] = fs::path(cam_names[i]).replace_extension(".tif").string();
  }

  for (size_t i = 0; i < cams.size(); i++) {

    // Check if we do a range
    if (skipCamera(i, opt)) continue;

    // Save the image using the block write function with multiple threads
    // Increase the tile size, as otherwise this code becomes very slow
    // since a time-consuming camera box calculation happens in each tile.
    vw::vw_out() << "Writing: " << image_names[i] << std::endl;
    bool has_georef = false; // the produced image is raw, it has no georef
    bool has_nodata = true;
    SatSimOptions local_opt = opt;
    local_opt.raster_tile_size = vw::Vector2i(1024, 1024);

    block_write_gdal_image(image_names[i], 
      vw::apply_mask(SynImageView(opt, cams[i], 
      dem_georef, dem, height_guess, ortho_georef, ortho, ortho_nodata_val), ortho_nodata_val),
      has_georef, ortho_georef,  // ortho georef will not be used, but needed for the api
      has_nodata, ortho_nodata_val, // borrow the nodata from ortho
      local_opt, vw::TerminalProgressCallback("", "\t--> "));
  }  

  // Write the list of images only if we are not skipping the first camera
  // Otherwise the same file may be written by multiple processes. Also
  // do this only for pinhole cameras, as for linescan there is only one 
  // image/camera rather than a set.
  if (opt.sensor_type == "pinhole") {
    if (!skipCamera(0, opt)) {
      std::string image_list = opt.out_prefix + "-images.txt"; 
      vw::vw_out() << "Writing: " << image_list << std::endl;
      asp::write_list(image_list, image_names);
    } else {
      // Print a warning message that the list won't be saved
      vw::vw_out(vw::WarningMessage) << "The image list is saved only when " 
        << "--first-index is 0, to avoid a race condition.\n";
    }
  }

  return;
}

} // end namespace asp
