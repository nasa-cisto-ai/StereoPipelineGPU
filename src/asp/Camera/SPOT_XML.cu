#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Core/Exception.h>          // for ArgumentErr, vw_throw, etc
#include <vw/Math/Quaternion.h>         // for Quat, Quaternion
#include <vw/Math/Vector.h>             // for Vector, Vector3, Vector4, etc
#include <vw/Cartography/Datum.h>       // for Datum
#include <vw/FileIO/DiskImageResourceGDAL.h>

#include <asp/Camera/XMLBase.h>
#include <asp/Camera/SPOT_XML.h>
#include <asp/Camera/RPCModel.h>
#include <asp/Camera/XMLBase.h>

#include <xercesc/parsers/XercesDOMParser.hpp>
#include <xercesc/sax/HandlerBase.hpp>
#include <xercesc/sax/SAXException.hpp>
#include <xercesc/sax/SAXParseException.hpp>
#include <xercesc/dom/DOMException.hpp>
#include <xercesc/util/PlatformUtils.hpp>
#include <xercesc/sax/ErrorHandler.hpp>

#include <boost/algorithm/string/predicate.hpp>
#include <boost/filesystem.hpp>
#include <boost/lexical_cast.hpp>
#include <iomanip>

using namespace vw;
using namespace vw::cartography;
using namespace xercesc;

using asp::XmlUtils::get_node;
using asp::XmlUtils::cast_xmlch;

namespace asp {


DOMElement* SpotXML::open_xml_file(std::string const& xml_path) {

  // Check if the file actually exists and throw a user helpful file.
  if ( !boost::filesystem::exists( xml_path ) )
    vw_throw( ArgumentErr() << "XML file \"" << xml_path << "\" does not exist." );

  std::string error_prefix = "XML file \"" + xml_path + "\" is invalid.\nException message is: \n";
  std::string err_message  = ""; // Filled in later on error

  try{
    //std::cout << "Set XML parser\n";
  
    // Set up the XML parser if we have not already done so
    if (!m_parser.get()) {
      m_parser.reset(new XercesDOMParser());
      m_errHandler.reset(new HandlerBase());
      m_parser->setValidationScheme(XercesDOMParser::Val_Always);
      m_parser->setDoNamespaces(true);   
      m_parser->setErrorHandler(m_errHandler.get());
    }

    //std::cout << "Load XML\n";

    // Load the XML file
    m_parser->parse( xml_path.c_str() );
    DOMDocument* xmlDoc      = m_parser->getDocument();
    DOMElement * elementRoot = xmlDoc->getDocumentElement();
    return elementRoot;

  } catch (const XMLException& toCatch) {
    char* message = XMLString::transcode(toCatch.getMessage());
    err_message = error_prefix + message;
    XMLString::release(&message);
  } catch (const DOMException& toCatch) {
    char* message = XMLString::transcode(toCatch.msg);
    err_message = error_prefix + message;
    XMLString::release(&message);
  } catch (const SAXException& toCatch) {
    char* message = XMLString::transcode(toCatch.getMessage());
    err_message = error_prefix + message;
    XMLString::release(&message);
  } catch ( const std::exception& e ) {
    err_message = error_prefix + e.what();
  } catch (...) {
    err_message = "Unrecognized error in XML file \"" + xml_path + "\"\n";
  }
  vw_throw( ArgumentErr() << err_message); // Only get here on error
  return 0;
}

void SpotXML::read_xml(std::string const& xml_path) {

  DOMElement * elementRoot = open_xml_file(xml_path);
  parse_xml(elementRoot);
}

std::vector<vw::Vector2> SpotXML::get_lonlat_corners(std::string const& xml_path) {
  SpotXML xml_reader;
  DOMElement * root = xml_reader.open_xml_file(xml_path);
  // Just get this one node we need to find the four corners
  DOMElement* raster_dims_node = get_node<DOMElement>(root, "Dataset_Frame");
  xml_reader.read_corners(raster_dims_node);
  return xml_reader.lonlat_corners;
}

void SpotXML::parse_xml(xercesc::DOMElement* node) {

  //std::cout << "Find dataset\n";
  xercesc::DOMElement* dataset_frame_node       = get_node<DOMElement>(node, "Dataset_Frame"); 
  //xercesc::DOMElement* crs_node                 get_node<DOMElement>(node, "Coordinate_Reference_System");
  //xercesc::DOMElement* image_display_node       get_node<DOMElement>(node, "Image_Display");
  //xercesc::DOMElement* scene_source_node        get_node<DOMElement>(node, "Scene_Source");
  //std::cout << "Find dims\n";
  xercesc::DOMElement* raster_dims_node         = get_node<DOMElement>(node, "Raster_Dimensions");
  xercesc::DOMElement* ephemeris_node           = get_node<DOMElement>(node, "Ephemeris");
  //std::cout << "Find ephem\n";
  xercesc::DOMElement* corrected_attitudes_node = get_node<DOMElement>(node, "Corrected_Attitudes");
  //std::cout << "Find angles\n";
  xercesc::DOMElement* look_angles_node         = get_node<DOMElement>(node, "Instrument_Look_Angles_List");
  xercesc::DOMElement* sensor_config_node       = get_node<DOMElement>(node, "Sensor_Configuration");
  

  //std::cout << "Parse dataset\n";
  read_corners(dataset_frame_node);
  //read_datum(crs_node);
  //read_display_info(image_display_node);
  //read_datetime(scene_source_node);
  //std::cout << "Parse dims\n";
  read_ephemeris(ephemeris_node);
  read_image_size(raster_dims_node);
  //std::cout << "Parse ephem\n";
  read_attitude(corrected_attitudes_node);
  //std::cout << "Parse angles\n";
  read_look_angles(look_angles_node);
  //std::cout << "Parse line times\n";
  read_line_times(sensor_config_node);
  
  // Set up the base time
  // - The position log starts before the image does, so the first
  //   time there should be a good reference time.
  boost::posix_time::ptime earliest_time = boost::posix_time::time_from_string("2016-05-04 00:00:00.00");
  std::list<std::pair<std::string, vw::Vector3> >::const_iterator iter;
  for (iter=position_logs.begin(); iter!=position_logs.end(); ++iter) {
    std::string s = iter->first;
    boost::replace_all(s, "T", " ");
    boost::posix_time::ptime this_time = boost::posix_time::time_from_string(s);
    if (this_time < earliest_time){
      earliest_time = this_time;
      //std::cout << "Using reference time " << iter->first << std::endl;
    }
  }
  m_time_ref_functor.set_base_time(earliest_time);
  //std::cout << "Done parsing XML.\n";
}


void SpotXML::read_look_angles(xercesc::DOMElement* look_angles_node) {

  // Set up the data storage
  const size_t num_cols = image_size.x();
  if (num_cols == 0)
    vw_throw(ArgumentErr() << "Did not load image size from SPOT XML file!\n");
  look_angles.resize(image_size.x());

  // Dig two levels down
  xercesc::DOMElement* look_angle_node
    = get_node<DOMElement>(look_angles_node, "Instrument_Look_Angles");
  xercesc::DOMElement* look_angle_list_node
    = get_node<DOMElement>(look_angle_node, "Look_Angles_List");

  // Pick out the "Angles" nodes
  DOMNodeList* children = look_angle_list_node->getChildNodes();

  size_t index = 0;
  const XMLSize_t num_children = children->getLength();
  for ( XMLSize_t i = 0; i < num_children; ++i ) {
    // Check child node type
    DOMNode* curr_node = children->item(i);
    if ( curr_node->getNodeType() != DOMNode::ELEMENT_NODE )
      continue;

    // Check the node name
    DOMElement* curr_element = dynamic_cast<DOMElement*>( curr_node );

    if (index >= num_cols)
      vw_throw(ArgumentErr() << "More look angles than rows in SPOT XML file!\n");

    // Look through the three nodes and assign each of them
    // - In this function we do this a little more by hand to try and speed things up
    DOMNodeList* sub_children = curr_element->getChildNodes();
    for ( XMLSize_t j = 0; j < sub_children->getLength(); ++j ) {

      DOMNode* child_node = sub_children->item(j);
      if ( child_node->getNodeType() != DOMNode::ELEMENT_NODE )
        continue;
      DOMElement* child_element = dynamic_cast<DOMElement*>( child_node );
      std::string tag2( XMLString::transcode(child_element->getTagName()) );
      std::string text( XMLString::transcode(child_element->getTextContent()) );

      if (tag2 == "DETECTOR_ID")
        look_angles[index].first = atoi(text.c_str());
      if (tag2 == "PSI_X")
        look_angles[index].second.x() = atof(text.c_str());
      if (tag2 == "PSI_Y")
        look_angles[index].second.y() = atof(text.c_str());
    }

    ++index;

  } // End loop through look angles
  if (index != num_cols)
    vw_throw(ArgumentErr() << "Did not load the correct number of SPOT5 pixel look angles!\n");
}

void SpotXML::read_ephemeris(xercesc::DOMElement* ephemeris_node) {

  position_logs.clear(); // Reset data storage
  velocity_logs.clear();

  // Dig one level down
  xercesc::DOMElement* points_node = get_node<DOMElement>(ephemeris_node, "Points");

  // Pick out the "Point" nodes
  DOMNodeList* children = points_node->getChildNodes();
  for ( XMLSize_t i = 0; i < children->getLength(); ++i ) {
    // Check child node type
    DOMNode* curr_node = children->item(i);
    if ( curr_node->getNodeType() != DOMNode::ELEMENT_NODE )
      continue;

    // Check the node name
    DOMElement* curr_element = dynamic_cast<DOMElement*>( curr_node );
    std::string tag( XMLString::transcode(curr_element->getTagName()) );
    if (tag.find("Point") == std::string::npos)
      continue;

    // Get the three sub-nodes
    xercesc::DOMElement* location_node = get_node<DOMElement>(curr_element, "Location");
    xercesc::DOMElement* velocity_node = get_node<DOMElement>(curr_element, "Velocity");
  
    // Read in both sets of values
    std::string time;
    Vector3 position, velocity;
    
    cast_xmlch( get_node<DOMElement>(curr_element, "TIME" )->getTextContent(), time );
    cast_xmlch( get_node<DOMElement>(location_node, "X")->getTextContent(), position.x() );
    cast_xmlch( get_node<DOMElement>(location_node, "Y")->getTextContent(), position.y() );
    cast_xmlch( get_node<DOMElement>(location_node, "Z")->getTextContent(), position.z() );
    cast_xmlch( get_node<DOMElement>(velocity_node, "X")->getTextContent(), velocity.x() );
    cast_xmlch( get_node<DOMElement>(velocity_node, "Y")->getTextContent(), velocity.y() );
    cast_xmlch( get_node<DOMElement>(velocity_node, "Z")->getTextContent(), velocity.z() );
    
    position_logs.push_back(std::pair<std::string, Vector3>(time, position));
    velocity_logs.push_back(std::pair<std::string, Vector3>(time, velocity));

  } // End loop through corrected attitudes
}



void SpotXML::read_attitude(xercesc::DOMElement* corrected_attitudes_node) {

  pose_logs.clear(); // Reset data storage

  // Dig one level down
  xercesc::DOMElement* corrected_attitude_node
    = get_node<DOMElement>(corrected_attitudes_node, "Corrected_Attitude");

  // Pick out the "Angles" nodes
  DOMNodeList* children = corrected_attitude_node->getChildNodes();
  for ( XMLSize_t i = 0; i < children->getLength(); ++i ) {
    // Check child node type
    DOMNode* curr_node = children->item(i);
    if ( curr_node->getNodeType() != DOMNode::ELEMENT_NODE )
      continue;

    // Check the node time
    DOMElement* curr_element = dynamic_cast<DOMElement*>( curr_node );
    std::string tag( XMLString::transcode(curr_element->getTagName()) );
    if (tag.find("Angles") == std::string::npos)
      continue;
  
    std::pair<std::string, Vector3> data;
    cast_xmlch( get_node<DOMElement>(curr_element, "YAW"  )->getTextContent(), data.second.x() );
    cast_xmlch( get_node<DOMElement>(curr_element, "PITCH")->getTextContent(), data.second.y() );
    cast_xmlch( get_node<DOMElement>(curr_element, "ROLL" )->getTextContent(), data.second.z() );
    cast_xmlch( get_node<DOMElement>(curr_element, "TIME" )->getTextContent(), data.first );
    pose_logs.push_back(data);

  } // End loop through corrected attitudes
}

void SpotXML::read_corners(xercesc::DOMElement* dataset_frame_node) {

  // Set up storage
  const size_t NUM_CORNERS = 4;
  lonlat_corners.resize(NUM_CORNERS);
  pixel_corners.resize(NUM_CORNERS);

  // Look through the four vertex nodes
  // - Currently we assume they are always in the same order!
  DOMNodeList* children = dataset_frame_node->getChildNodes();
  size_t count = 0;
  for ( XMLSize_t i = 0; i < children->getLength(); ++i ) {
    // Check child node type
    DOMNode* curr_node = children->item(i);
    if ( curr_node->getNodeType() != DOMNode::ELEMENT_NODE )
      continue;

    // Check the node time
    DOMElement* curr_element = dynamic_cast<DOMElement*>( curr_node );

    // There should only be four vertex nodes here.
    // Parse the values.
    XmlUtils::cast_xmlch(XmlUtils::get_node<DOMElement>(curr_element, "FRAME_LON")->getTextContent(), lonlat_corners[count].x());
    XmlUtils::cast_xmlch(XmlUtils::get_node<DOMElement>(curr_element, "FRAME_LAT")->getTextContent(), lonlat_corners[count].y());
    XmlUtils::cast_xmlch(XmlUtils::get_node<DOMElement>(curr_element, "FRAME_ROW")->getTextContent(), pixel_corners[count].y());
    XmlUtils::cast_xmlch(XmlUtils::get_node<DOMElement>(curr_element, "FRAME_COL")->getTextContent(), pixel_corners[count].x());
    ++count;
    if (count == NUM_CORNERS)
      return;

  } // End loop through the vertex nodes
}


void SpotXML::read_image_size(xercesc::DOMElement* raster_dims_node) {
  cast_xmlch( get_node<DOMElement>(raster_dims_node, "NROWS")->getTextContent(), image_size[1] );
  cast_xmlch( get_node<DOMElement>(raster_dims_node, "NCOLS")->getTextContent(), image_size[0] );
}

void SpotXML::read_line_times(xercesc::DOMElement* sensor_config_node) {
  cast_xmlch( get_node<DOMElement>(sensor_config_node, "LINE_PERIOD"      )->getTextContent(), line_period);
  cast_xmlch( get_node<DOMElement>(sensor_config_node, "SCENE_CENTER_TIME")->getTextContent(), center_time);
  cast_xmlch( get_node<DOMElement>(sensor_config_node, "SCENE_CENTER_LINE")->getTextContent(), center_line);
  cast_xmlch( get_node<DOMElement>(sensor_config_node, "SCENE_CENTER_COL" )->getTextContent(), center_col);
  center_line -= 1;
  center_col  -= 1; // Convert from 1-based to 0-based indices.
}

// ----- These functions help convert the input data to a useable format ------

// Input strings look like this: 2008-03-04T12:31:03.081912
double SpotXML::convert_time(std::string const& s) const {
  try{
    // Replace the T with a space so the default Boost function can parse the time.
    std::string s2 = s;
    boost::replace_all(s2, "T", " ");
    boost::posix_time::ptime time = boost::posix_time::time_from_string(s2);
    return this->m_time_ref_functor(time);
  }catch(...){
    vw::vw_throw(vw::ArgumentErr() << "Failed to parse time from string: " << s << "\n");
  }
  return -1; // Never reached!
}

// This is pretty simple, SPOT5 has a constant time for each line.
vw::camera::LinearTimeInterpolation SpotXML::setup_time_func() const {

  // The metadata tells us the time of the middle line, so find the time for the first line.
  double center_time_d = convert_time(this->center_time);
  double min_line_diff = static_cast<double>(0 - this->center_line);
  double min_line_time = center_time_d + this->line_period*min_line_diff;
  //std::cout << "Setup time functor: " << std::setprecision(12)  << min_line_time << ", " << this->line_period << std::endl;
  //std::cout << std::setprecision(12)  << "Center time: " << center_time_d << std::endl;
  return vw::camera::LinearTimeInterpolation(min_line_time, this->line_period);
}


// Velocities are the sum of inertial velocities and the instantaneous
//  Earth rotation.

// The velocity is already in GCC, so just pack into a function.
vw::camera::LagrangianInterpolation SpotXML::setup_velocity_func() const {

  const int INTERP_RADII = 4; // Reccomended in the docs
  std::vector<double>  time;
  std::vector<Vector3> velocity;

  // Loop through the velocity logs and extract values
  std::list<std::pair<std::string, vw::Vector3> >::const_iterator iter;
  for (iter=velocity_logs.begin(); iter!=velocity_logs.end(); ++iter) {
    time.push_back(convert_time(iter->first));
    velocity.push_back(iter->second);
    //std::cout << "Adding velocity point: " << iter->first 
    //          << " --> " << iter->second << std::endl;
  }
  
  // More generic method for variable time intervals
  //return vw::camera::LagrangianInterpolationVarTime(velocity, time, INTERP_RADII);
  
  // A faster method for when we know the time delta is constant
  double min_time   = time[0];
  double max_time   = time[time.size()-1];
  double time_delta = (max_time - min_time) / (time.size()-1);
  return vw::camera::LagrangianInterpolation(velocity, min_time, time_delta, max_time, INTERP_RADII);
}

// The position is already in GCC, so just pack into a function.
// - Currently this is identical to the velocity function, but this may change later.
vw::camera::LagrangianInterpolation SpotXML::setup_position_func() const {

 const int INTERP_RADII = 4; // Reccomended in the docs
  std::vector<double>  time;
  std::vector<Vector3> position;

  // Loop through the velocity logs and extract values
  std::list<std::pair<std::string, vw::Vector3> >::const_iterator iter;
  for (iter=position_logs.begin(); iter!=position_logs.end(); ++iter) {
    time.push_back(convert_time(iter->first));
    position.push_back(iter->second);
    //std::cout << "Adding position point: " << convert_time(iter->first)
    //          << " --> " << iter->second << std::endl;
  }
  
  // More generic method for variable time intervals
  //return vw::camera::LagrangianInterpolationVarTime(position, time, INTERP_RADII);
  
  // A faster method for when we know the time delta is constant
  double min_time   = time[0];
  double max_time   = time[time.size()-1];
  double time_delta = (max_time - min_time) / (time.size()-1);
  return vw::camera::LagrangianInterpolation(position, min_time, time_delta, max_time, INTERP_RADII);
}

vw::camera::LinearPiecewisePositionInterpolation SpotXML::setup_pose_func(
        vw::camera::LinearTimeInterpolation const& time_func) const {

  // This function returns a functor that returns just the yaw/pitch/roll angles.
  // - The time interval between lines is not constant but it is extremely close.


  // For some reason the corrected pose angles do not start early enough to cover
  // the time span for all of the input lines!
  // - In order to handle this, we repeat the earliest pose value so that it starts
  //   before the first line.
  // - The raw pose angles do start before the lines, but their values differ noticably
  //   from the corrected values.
  
  // Compute how many padded pose entries are needed to cover all of the lines.
  size_t num_lines           = this->image_size[1];
  double num_corrected_poses = static_cast<double>(pose_logs.size());
  double first_line_time     = time_func(0);
  double last_line_time      = time_func(num_lines - 1.0);
  double pose_start_time     = convert_time(pose_logs.front().first);
  double pose_stop_time      = convert_time(pose_logs.back().first);
  double pose_delta_t        = (pose_stop_time - pose_start_time) / (num_corrected_poses - 1.0);
  int    num_prefill_poses   = static_cast<int>(ceil((pose_start_time - first_line_time) / pose_delta_t));
  int    num_postfill_poses  = static_cast<int>(ceil((last_line_time  - pose_stop_time ) / pose_delta_t));
  //std::cout << "First line time: " << first_line_time << std::endl;
  //std::cout << "Last line time:  " << last_line_time  << std::endl;
  //std::cout << "Pose start: " << pose_start_time << std::endl;
  //std::cout << "Pose stop:  " << pose_stop_time  << std::endl;
  num_postfill_poses += 1; // Stick another bit of padding on the back.
                           // This is so our Extrinsics algorithms have enough room to interpolate.
  if (num_prefill_poses < 1)
    num_prefill_poses = 0;
  if (num_postfill_poses < 1)
    num_postfill_poses = 0;
    
  size_t num_total_poses = pose_logs.size() + static_cast<size_t>(num_prefill_poses )
                                            + static_cast<size_t>(num_postfill_poses);

  std::vector<Vector3> pose(num_total_poses);
  std::vector<double>  time(num_total_poses);
  
  // Fill in the pre-padding poses
  size_t index = 0;
  for (int i=0; i<num_prefill_poses; ++i) {
    double time_offset = pose_delta_t*static_cast<double>(num_prefill_poses-i);
    time[index] = convert_time(pose_logs.front().first) - time_offset;
    pose[index] = pose_logs.front().second;
    //std::cout << "PREFILL: " << time[index] << std::endl;
    ++index;
  }

  // Now fill in the real poses
  std::list<std::pair<std::string, vw::Vector3> >::const_iterator iter;
  for (iter=pose_logs.begin(); iter!=pose_logs.end(); ++iter) {
    time[index] = convert_time(iter->first);
    pose[index] = iter->second;
    ++index;
  }

  // Fill in the post-padding poses
  for (int i=0; i<num_postfill_poses; ++i) {
    double time_offset = pose_delta_t*(i+1);
    time[index] = convert_time(pose_logs.back().first) + time_offset;
    pose[index] = pose_logs.back().second;
    //std::cout << "POSTFILL: " << time[index] << std::endl;
    ++index;
  }
  
  //double max_time = time.back();
  double min_time = time.front();
  
  //std::cout << std::setprecision(12) << "Adding pose info: " << min_time << ", " 
  //          << max_time << " -> " << pose_delta_t << std::endl;
  
  return vw::camera::LinearPiecewisePositionInterpolation(pose, min_time, pose_delta_t);

}


} // end namespace asp


