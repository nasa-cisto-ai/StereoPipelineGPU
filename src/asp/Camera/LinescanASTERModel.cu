#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <asp/Camera/ASTER_XML.h>
#include <vw/Camera/CameraSolve.h>
#include <asp/Camera/LinescanASTERModel.h>
namespace asp {

using namespace vw;

ASTERCameraModel::ASTERCameraModel(std::vector< std::vector<vw::Vector2> > const& lattice_mat,
				   std::vector< std::vector<vw::Vector3> > const& sight_mat,
				   std::vector< std::vector<vw::Vector3> > const& world_sight_mat,
				   std::vector<vw::Vector3>                const& sat_pos,
				   vw::Vector2                             const& image_size,
				   boost::shared_ptr<vw::camera::CameraModel> rpc_model):
  m_lattice_mat(lattice_mat), m_sight_mat(sight_mat),
  m_world_sight_mat(world_sight_mat),
  m_sat_pos(sat_pos), m_image_size(image_size), m_rpc_model(rpc_model){

  if (m_lattice_mat.empty() || m_lattice_mat[0].empty()) 
    vw::vw_throw( vw::ArgumentErr() << "Empty matrix of lattice points.\n" );
  
  int min_col = m_lattice_mat.front().front().x();
  int min_row = m_lattice_mat.front().front().y();
  
  int max_col = m_lattice_mat.back().back().x();
  int max_row = m_lattice_mat.back().back().y();
  
  int num_rows = m_lattice_mat.size();
  int num_cols = m_lattice_mat.front().size();
      
  // The spacing between rows must be integer
  double tol = 1e-10;
  double d_row = double(max_row - min_row)/(num_rows - 1.0);
  if (std::abs(d_row - round(d_row)) > tol) {
    vw::vw_throw( vw::ArgumentErr()
                  << "The spacing between lattice points must be integer.\n" );
  }
  d_row = round(d_row);

  // The spacing between columns must be integer
  double d_col = double(max_col - min_col)/(num_cols - 1.0);
  if (std::abs(d_col - round(d_col)) > tol) {
    vw::vw_throw( vw::ArgumentErr()
                  << "The spacing between lattice points must be integer.\n" );
  }
  d_col = round(d_col);

  if ((int)m_sat_pos.size() != num_rows) {
    vw::vw_throw( vw::ArgumentErr()
                  << "The number of rows of lattice points does not "
                  << "agree with the number of satellite positions.\n" );
  }

  m_interp_sat_pos
    = vw::camera::LinearPiecewisePositionInterpolation(m_sat_pos, min_row, d_row);

  m_interp_sight_mat
    = vw::camera::SlerpGridPointingInterpolation(m_world_sight_mat, min_row, d_row, min_col, d_col);

#if 0
  // This is useful in testing how well point_to_pixel() works for given point and pixel.
  double spacing = 9.5655;
  double max_err = 0;
  for (double col = 0; col < m_image_size[0]; col+= spacing) {
    for (double row = 0; row < m_image_size[1]; row+= spacing) {
      Vector2 pix(col, row);

      double datum_h = 6378137;
      double h = datum_h + 1000; // 1000 m above the datum
      Vector3 C = this->camera_center(pix);
      Vector3 D = this->pixel_to_vector(pix);

      double Delta = dot_prod(C, D)*dot_prod(C, D) - dot_prod(D, D)*(dot_prod(C, C) - h*h);
      double t = ( -dot_prod(C, D) - sqrt(Delta) ) / dot_prod(D, D);
      Vector3 P = this->camera_center(pix) + t * this->pixel_to_vector(pix);

      std::cout << "cam ctr is in km: " << norm_2(this->camera_center(pix))/1000 << std::endl;
      std::cout << "point height and diff in km: "  << norm_2(P)/1000  << ' '
		<< (norm_2(P) - 6378137)/1000 << std::endl;

      Vector2 pix2 = this->point_to_pixel(P);
      double err = norm_2(pix-pix2);
      std::cout << "Pixel error: " << pix << ' ' << pix2 << ' ' << err << std::endl;
      max_err = std::max(err, max_err);
    }
  }
  std::cout << "max err is " << max_err << std::endl;
#endif
}

// Project the point onto the camera. Sometimes, but not always, seeding with the RPC
// model is beneficial.  
vw::Vector2 ASTERCameraModel::point_to_pixel(Vector3 const& point, Vector2 const& start_in) const {

  // - This method will be slower but works for more complicated geometries
  vw::camera::CameraGenericLMA model( this, point );
  int status;
  vw::Vector2 start = m_image_size / 2.0; // Use the center as the initial guess

  bool has_guess = false;
  
  // If the user provided a column number guess.
  if (start_in[0] >= 0) {
    start[0] = start_in[0];
    has_guess = true;
  }
  
  // If the user provided a row number guess.
  if (start_in[1] >= 0) {
    start[1] = start_in[1];
    has_guess = true;
  }

  if (!has_guess) {
    double min_err = norm_2(model(start));
    // No good initial guess. The method will fail to converge.
    // Iterate through the lattice to find a good initial guess.
    for (int row = 0; row < int(m_lattice_mat.size())-1; row++) {
      // TODO: Experiment more with the number below.
      int T = 100; // This way we'll sample about every 4-th pixel since dcol = 400
      int col = m_lattice_mat.front().size()/2;
      for (int r = 0; r < T; r++) {
	double wr = double(r)/(T-1.0);
	Vector2 pt
	  = wr*m_lattice_mat[row+1][col]
	  + (1-wr)*m_lattice_mat[row][col];
	double err = norm_2(model(pt));
	if (err < min_err) {
	  min_err = err;
	  start = pt;
	}
      }
    }
  }

#if 0
  // This exhaustive search for an initial guess is an overkill
  if (!has_guess) {
    double min_err = norm_2(model(start));
    // No good initial guess. The method will fail to converge.
    // Iterate through the lattice to find a good initial guess.
    for (int row = 0; row < int(m_lattice_mat.size())-1; row++) {
      for (int col = 0; col < int(m_lattice_mat.front().size())-1; col++) {
	int T = 100;
	for (int r = 0; r < T; r++) {
	  for (int c = 0; c < T; c++) {
	    double wr = double(r)/(T-1.0);
	    double wc = double(c)/(T-1.0);
	    Vector2 pt
	      = wr*wc*m_lattice_mat[row+1][col+1]
	      + (1-wr)*wc*m_lattice_mat[row][col+1]
	      + wr*(1-wc)*m_lattice_mat[row+1][col]
	      + (1-wr)*(1-wc)*m_lattice_mat[row][col];
	    double err = norm_2(model(pt));
	    if (err < min_err) {
	      min_err = err;
	      start = pt;
	    }
	  }
	}
      }
    }
  }
#endif

  // Solver constants
  const double ABS_TOL = 1e-16;
  const double REL_TOL = 1e-16;
  const int    MAX_ITERATIONS = 1e+5;
  const double MAX_ERROR = 1e-2;

  // Try two initial guesses. TODO: Study this in more detail.
  
  // Solution with user-provided initial guess
  Vector3 objective(0, 0, 0);
  vw::Vector2 solution1 = vw::math::levenberg_marquardtFixed<vw::camera::CameraGenericLMA, 2,3>
    (model, start, objective, status, ABS_TOL, REL_TOL, MAX_ITERATIONS);
  
  // Solution with the RPC initial guess
  Vector2 start_rpc = this->m_rpc_model->point_to_pixel(point);
  vw::Vector2 solution2 = vw::math::levenberg_marquardtFixed<vw::camera::CameraGenericLMA, 2,3>
    (model, start_rpc, objective, status, ABS_TOL, REL_TOL, MAX_ITERATIONS);
  
  double error1 = norm_2(model(solution1));
  double error2 = norm_2(model(solution2));
  double error  = std::min(error1, error2);
  
  vw::Vector2 solution;
  if (error1 < error2) {
    solution = solution1;
  } else if (error1 > error2) {
    solution = solution2;
  }
  
  // Check the error - If it is too high then the solver probably got
  // stuck at the edge of the image.
  VW_ASSERT( (status > 0) && (error < MAX_ERROR),
             vw::camera::PointToPixelErr() << "Unable to project point into LinescanASTER model" );
  
  return solution;
}

vw::Vector2 ASTERCameraModel::point_to_pixel(Vector3 const& point, double starty) const {
  return this->point_to_pixel(point, Vector2(-1.0, starty));
}
  
vw::Vector3 ASTERCameraModel::camera_center(vw::Vector2 const& pix) const {
  return m_interp_sat_pos(pix.y());
}
    
vw::Vector3 ASTERCameraModel::pixel_to_vector(vw::Vector2 const& pixel) const{
  try {
    return m_interp_sight_mat(pixel);
  } catch(const vw::Exception &e) {
    // Repackage any of our exceptions thrown below this point as a 
    //  pixel to ray exception that other code will be able to handle.
    vw_throw(vw::camera::PixelToRayErr() << e.what());
  }
  return vw::Vector3(); // Never reached
}
    
boost::shared_ptr<ASTERCameraModel>
load_ASTER_camera_model_from_xml(std::string const& path,
                                 boost::shared_ptr<vw::camera::CameraModel> rpc_model){
  
  // XYZ coordinates are in the ITRF coordinate frame which means GCC coordinates.
  // - The velocities are in the same coordinate frame, not in some local frame.
  
  vw_out(vw::DebugMessage,"asp") << "Loading ASTER camera file: " << path << std::endl;
  
  // Parse the ASTER XML file
  ASTERXML xml_reader;
  xml_reader.read_xml(path);

  // Feed everything into a new camera model.
  return boost::shared_ptr<ASTERCameraModel>(new ASTERCameraModel(xml_reader.m_lattice_mat,
								  xml_reader.m_sight_mat,
								  xml_reader.m_world_sight_mat,
								  xml_reader.m_sat_pos,
								  xml_reader.m_image_size,
								  rpc_model));
} // End function load_ASTER_camera_model()


} // end namespace asp

