#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Logic for propagation of errors (covariance) through stereo triangulation 

#include <asp/Camera/Covariance.h>
#include <asp/Camera/LinescanDGModel.h>
#include <asp/Core/StereoSettings.h>

#include <vw/Stereo/StereoModel.h>
#include <vw/Math/LinearAlgebra.h>

#include <iostream>

using namespace vw::camera;

namespace asp {

// Change in satellite position (measured in meters) and satellite orientation
// measured in normalized quaternions, to be used for numerical differencing.
// One has to be mindful of the fact that the positions are on the order of
// 7.0e6 meters given the distance from satellite to Earth center in ECEF,
// so the position delta should not be too tiny.
const double deltaPosition = 0.01; // measured in meters
const double deltaQuat     = 1.0e-6; // given that quaternions are normalized

// Given 0 <= num < 15, return a perturbation in satellite position. The
// starting one is the zero perturbation, then perturb first
// coordinate in the positive and then negative direction, then same
// for second and third coordinate. The rest of the perturbations are
// 0 as those indices are used to perturb the quaternions.
// So, return (0, 0, 0), (deltaPosition, 0, 0), (-deltaPosition, 0, 0)
// (0, deltaPosition, 0), (0, -deltaPosition, 0), and so on.
vw::Vector<double, 3> positionDelta(int num) {

  vw::Vector<double, 3> ans; // this is 0
  if (num == 0)
    return ans; // nominal position value, no perturbation
  if (num > 6) 
    return ans; // nominal position value, will perturb the quaternion then

  double sign = 1.0; // sign of the perturbation: 1, -1, 1, -1, etc.
  if (num % 2 == 0) 
    sign = -1.0;

  int index = (num - 1)/2; // get 0, 1, or 2
  ans[index] = sign * deltaPosition;
  
  return ans;
}

// Similar logic as above for the quaternion, for indices 7, ..., 14 (8 of them)
vw::Vector<double, 4> quatDelta(int num) {

  vw::Vector<double, 4> ans; // this is 0
  if (num <= 6) 
    return ans; // nominal quat value, will perturb the positions then

  num = num - 6; // now num = 1, 2, ... 8
  if (num > 8) 
    vw::vw_throw(vw::ArgumentErr() << "Out of bounds in quatDelta().\n");
  
  double sign = 1.0; // sign of the perturbation: 1, -1, 1, -1, etc.
  if (num % 2 == 0) 
    sign = -1.0;

  int index = (num - 1)/2; // get 0, 1, 2, or 3
  ans[index] = sign * deltaQuat;
  
  return ans;
}

// Number of nominal and perturbed cameras when the covariance is computed with DG cameras.
int numCamsForCovariance() {
  // One nominal camera. Then one positive and negative perturbation
  // for each position (3) and quaternion (4).
  return 15; 
}

// Given two DG cameras and a pixel in each camera image, consider the
// following transform. Go from the perturbed joint vector of
// satellite positions and quaternions for this pixel pair to the
// perturbed triangulated point. Then, the vector from nominal to
// perturbed triangulation point is converted to North-East-Down
// relative to the nominal point. Use numerical differentiation to
// find the Jacobian of this transform with centered
// differences. This will be used to find the covariances of the
// triangulated point in NED coordinates given the input satellite
// covariances. This works only for Maxar (DigitalGlobe)
// cameras. This function may throw exceptions. Do not divide the
// numerical derivatives by deltaPosition and deltaQuat, but only by
// 2.0 (since these are centered differences). That because the
// division makes the partial derivatives in quaternions huge and is
// not good for numerical stability. We will compensate for this
// when we multiply by the actual covariances, which are huge, so
// those will be pre-multiplied by the squares of deltaPosition and
// deltaQuat, with the same final result.
void scaledDGTriangulationJacobian(vw::cartography::Datum const& datum,
                                   vw::camera::CameraModel const* cam1,
                                   vw::camera::CameraModel const* cam2,
                                   vw::Vector2 const& pix1,
                                   vw::Vector2 const& pix2,
                                   vw::Matrix<double> & J) {
  
  // Handle adjusted cameras
  bool adjusted_cameras = false;
  const AdjustedCameraModel *adj_cam1 = dynamic_cast<const AdjustedCameraModel*>(cam1);
  const AdjustedCameraModel *adj_cam2 = dynamic_cast<const AdjustedCameraModel*>(cam2);
  if ((adj_cam1 == NULL) != (adj_cam2 == NULL))
    vw::vw_throw(vw::ArgumentErr() << "The cameras must be either both "
                 << "adjusted or both unadjusted.\n");

  vw::Matrix3x3 cam1_rot, cam2_rot;
  vw::Vector3 cam1_shift, cam2_shift;
  if (adj_cam1 != NULL && adj_cam2 != NULL) {
    adjusted_cameras = true;
    // transforms from unadjusted to adjusted coordinates
    vw::Matrix4x4 cam1_adj = adj_cam1->ecef_transform();
    vw::Matrix4x4 cam2_adj = adj_cam2->ecef_transform();
    cam1_rot = submatrix(cam1_adj, 0, 0, 3, 3);
    cam2_rot = submatrix(cam2_adj, 0, 0, 3, 3);
    cam1_shift = vw::Vector3(cam1_adj(0, 3), cam1_adj(1, 3), cam1_adj(2, 3));
    cam2_shift = vw::Vector3(cam2_adj(0, 3), cam2_adj(1, 3), cam2_adj(2, 3));
  }
  
  DGCameraModel const* dg_cam1 = dynamic_cast<DGCameraModel const*>(unadjusted_model(cam1));
  DGCameraModel const* dg_cam2 = dynamic_cast<DGCameraModel const*>(unadjusted_model(cam2));
  if (dg_cam1 == NULL || dg_cam2 == NULL) 
    vw::vw_throw(vw::ArgumentErr() << "Expecting DG cameras.\n");

  // Numerical differences will be used. Camera models with deltaPosition and deltaQuat
  // perturbations have already been created in LinescanDGModel.cc using the positionDelta()
  // and quatDelta() functions from above.
  if (dg_cam1->m_perturbed_cams.empty() || dg_cam2->m_perturbed_cams.empty()) 
    vw::vw_throw(vw::ArgumentErr() << "The perturbed cameras were not set up.\n");
  
  if (dg_cam1->m_perturbed_cams.size() != dg_cam2->m_perturbed_cams.size())
    vw::vw_throw(vw::ArgumentErr()
                 << "The number of perturbations in the two cameras do not agree.\n");

  // Find the camera center and direction for first unperturbed
  // camera, and for the perturbed versions. Same for the second
  // camera.
  std::vector<vw::Vector3> cam1_dirs, cam1_ctrs, cam2_dirs, cam2_ctrs;
  cam1_dirs.push_back(dg_cam1->pixel_to_vector(pix1));
  cam1_ctrs.push_back(dg_cam1->camera_center(pix1));
  cam2_dirs.push_back(dg_cam2->pixel_to_vector(pix2));
  cam2_ctrs.push_back(dg_cam2->camera_center(pix2));
  for (size_t it = 0; it < dg_cam1->m_perturbed_cams.size(); it++) {
    cam1_dirs.push_back(dg_cam1->m_perturbed_cams[it]->pixel_to_vector(pix1));
    cam1_ctrs.push_back(dg_cam1->m_perturbed_cams[it]->camera_center(pix1));
    cam2_dirs.push_back(dg_cam2->m_perturbed_cams[it]->pixel_to_vector(pix2));
    cam2_ctrs.push_back(dg_cam2->m_perturbed_cams[it]->camera_center(pix2));
  }

  // Apply adjustments
  if (adjusted_cameras) {
    for (size_t it = 0; it < cam1_dirs.size(); it++) {
      cam1_dirs[it] = cam1_rot * cam1_dirs[it];
      cam2_dirs[it] = cam2_rot * cam2_dirs[it];
      cam1_ctrs[it] = cam1_rot * cam1_ctrs[it] + cam1_shift;
      cam2_ctrs[it] = cam2_rot * cam2_ctrs[it] + cam2_shift;
    }
  }
  
  // Nominal triangulation point
  vw::Vector3 tri_nominal, err_nominal;
  // If triangulation fails, it can return NaN
  tri_nominal
    = vw::stereo::triangulate_pair(cam1_dirs[0], cam1_ctrs[0], cam2_dirs[0], cam2_ctrs[0],
                                   err_nominal);
  if (tri_nominal != tri_nominal) // NaN
    vw::vw_throw(vw::ArgumentErr() << "Could not triangulate.\n");

  // The matrix to go from the NED coordinate system to ECEF
  vw::Vector3 llh = datum.cartesian_to_geodetic(tri_nominal);
  vw::Matrix3x3 NedToEcef = datum.lonlat_to_ned_matrix(subvector(llh, 0, 2));
  vw::Matrix3x3 EcefToNed = inverse(NedToEcef);

  // There are 14 input variables: 3 positions and 4 quaternions for
  // cam1, and same for cam2. For each of them must compute a centered
  // difference. The output has 3 variables. As documented above,
  // the vector from the nominal to perturbed
  // triangulated point will be converted to North-East-Down
  // coordinates at the nominal triangulated point.
  J.set_size(3, 14);
  J.set_zero();
  for (int coord = 0; coord < 14; coord++) {

    vw::Vector3 cam1_dir_plus, cam1_ctr_plus, cam2_dir_plus, cam2_ctr_plus;
    vw::Vector3 cam1_dir_minus, cam1_ctr_minus, cam2_dir_minus, cam2_ctr_minus;
    if (coord < 7) {
      // The perturbed cameras store positive and negative
      // perturbations, in alternating order. See positionDelta() and
      // quatDelta() for the book-keeping. Note that a perturbation in
      // the satellite quaternion also affects the camera center,
      // given how one converts from satellite to camera coordinates
      // when the DG model is created.

      // Since at position 0 in cam_dirs we store the nominal (unperturbed)
      // values, add 1 below.
      cam1_dir_plus  = cam1_dirs[2*coord + 1]; cam1_ctr_plus  = cam1_ctrs[2*coord + 1];
      cam1_dir_minus = cam1_dirs[2*coord + 2]; cam1_ctr_minus = cam1_ctrs[2*coord + 2];
    } else {
      // When variables affecting second camera change, the first one stays at nominal value.
      cam1_dir_plus  = cam1_dirs[0]; cam1_ctr_plus  = cam1_ctrs[0];
      cam1_dir_minus = cam1_dirs[0]; cam1_ctr_minus = cam1_ctrs[0];
    }

    // For the second camera, the book-keeping is reversed
    if (coord < 7) {
      // Second camera values do not change when first camera inputs change
      cam2_dir_plus  = cam2_dirs[0]; cam2_ctr_plus  = cam2_ctrs[0];
      cam2_dir_minus = cam2_dirs[0]; cam2_ctr_minus = cam2_ctrs[0];
    } else {
      int coord2 = coord - 7; // has values 0, 1, ..., 6
      cam2_dir_plus  = cam2_dirs[2*coord2 + 1]; cam2_ctr_plus  = cam2_ctrs[2*coord2 + 1];
      cam2_dir_minus = cam2_dirs[2*coord2 + 2]; cam2_ctr_minus = cam2_ctrs[2*coord2 + 2];
    }

    vw::Vector3 tri_plus, err_plus, tri_minus, err_minus;
    tri_plus = vw::stereo::triangulate_pair(cam1_dir_plus, cam1_ctr_plus,
                                            cam2_dir_plus, cam2_ctr_plus, err_plus);
    tri_minus = vw::stereo::triangulate_pair(cam1_dir_minus, cam1_ctr_minus,
                                             cam2_dir_minus, cam2_ctr_minus, err_minus);

    // Find the triangulated points in the local NED (horizontal-vertical)
    // coordinate system.
    vw::Vector3 ned_plus = EcefToNed * (tri_plus - tri_nominal);
    vw::Vector3 ned_minus = EcefToNed * (tri_minus - tri_nominal);

    // Find the numerical partial derivative, but do not divide by the
    // spacing (deltaPosition or deltaQuat) as that makes the numbers
    // huge. We will compensate for when use use this Jacobian to
    // propagate the satellite position and quaternion covariances
    // (matrix SC), by the formula J * SC * J^T. Then, we will divide SC
    // by these squared delta quantities, which is the right thing to
    // do, because the values in SC are tiny, and, in fact, on the
    // order of the squares of the delta values.
    vw::Vector3 ned_diff = (ned_plus - ned_minus)/2.0;
    
    for (int row = 0; row < 3; row++) 
      J(row, coord) = ned_diff[row];
  }

  return;
}

// Given upper-right values in a symmetric matrix of given size, find
// the lower-left values by reflection, and insert them as a block
// starting at the desired row and column. Used to populate the joint
// covariance matrix. Per DigitalGlobe's doc, the covariances are
// stored as c11, c12, c13, ..., c22, c23, ...
void insertBlock(int start, int size, double* inputVals, vw::Matrix<double> & C) {
  int count = 0;
  for (int row = 0; row < size; row++) {
    for (int col = row; col < size; col++) {
      C(start + row, start + col) = inputVals[count];
      C(start + col, start + row) = inputVals[count];
      count++;
    }
  }
}
  
// Based on tabulated satellite position and quaternion covariance
// for each DG camera, find the interpolated covariances for cam1 at
// pix1 (6 for position, 10 for orientation, as just the upper-right
// corner is used), same for cam2 at pix2, autocomplete these to the
// full matrices (3x3 and 4x4 for each), create a combined matrix of
// covariances (14 x 14), and divide the entries by squares of
// deltaPosition and deltaQuat which normalizes them, and which are
// compensated by not dividing by these numbers (without the square)
// what is found in scaledDGTriangulationJacobian(). Later we will do
// J * C * J^T. The same order of variables as in
// scaledDGTriangulationJacobian must be used.
void scaledDGSatelliteCovariance(vw::camera::CameraModel const* cam1,
                                 vw::camera::CameraModel const* cam2,
                                 vw::Vector2 const& pix1,
                                 vw::Vector2 const& pix2,
                                 vw::Matrix<double> & C) {
  
  // Initialize the output
  // 3 positions for cam 1, 4 orientations for cam1, 3 positions for cam2, 4 orientations
  // for cam2. So, four blocks in total. The resulting matrix must be symmetric.
  C.set_size(14, 14);
  C.set_zero();

  // Here it is not important that the camera are adjusted or not, as all that is needed
  // are the input covariances.
  DGCameraModel const* dg_cam1 = dynamic_cast<DGCameraModel const*>(unadjusted_model(cam1));
  DGCameraModel const* dg_cam2 = dynamic_cast<DGCameraModel const*>(unadjusted_model(cam2));
  if (dg_cam1 == NULL || dg_cam2 == NULL) 
    vw::vw_throw(vw::ArgumentErr() << "Expecting DG cameras.\n");

  // Find the covariances at given pixel by interpolation in the input table.
  // Use nearest neighbor interpolation as covariances are known with
  // just a few digits of precision and are not meant to be smooth.
  double p_cov1[SAT_POS_COV_SIZE], p_cov2[SAT_POS_COV_SIZE];
  double q_cov1[SAT_QUAT_COV_SIZE], q_cov2[SAT_QUAT_COV_SIZE];  
  dg_cam1->interpSatellitePosCov(pix1, p_cov1);
  dg_cam1->interpSatelliteQuatCov(pix1, q_cov1);
  dg_cam2->interpSatellitePosCov(pix2, p_cov2);
  dg_cam2->interpSatelliteQuatCov(pix2, q_cov2);

  // This is useful for seeing which input covariance has a bigger effect.
  // The default value of these factors is 1.
  double pf = asp::stereo_settings().position_covariance_factor;
  double qf = asp::stereo_settings().orientation_covariance_factor;
  
  // Scale these per scaledDGTriangulationJacobian().
  for (int ip = 0; ip < SAT_POS_COV_SIZE; ip++) {
    p_cov1[ip] = pf * p_cov1[ip] / (deltaPosition * deltaPosition); 
    p_cov2[ip] = pf * p_cov2[ip] / (deltaPosition * deltaPosition); 
  }
  for (int iq = 0; iq < SAT_QUAT_COV_SIZE; iq++) {
    q_cov1[iq] = qf * q_cov1[iq] / (deltaQuat * deltaQuat); 
    q_cov2[iq] = qf * q_cov2[iq] / (deltaQuat * deltaQuat); 
  }

  // Put these in the covariance matrix
  insertBlock(0,  3, p_cov1, C);
  insertBlock(3,  4, q_cov1, C);
  insertBlock(7,  3, p_cov2, C);
  insertBlock(10, 4, q_cov2, C);

#if 0
  std::cout << "Produced matrix " << std::endl;
  for (int row = 0; row < 14; row++) {
    for (int col = 0; col < 14; col++) {
      std::cout << C(row, col) << " ";
    }
    std::cout << std::endl;
  }

  // Debug code. This shows that some quaternion covariances have a
  // negative determinant. That is because an eigenvalue is very close
  // to 0 or even negative (but small). This singularity goes away
  // after the covariances are propagated.
  std::cout << "determinant1 " << det(submatrix(C, 0, 0, 3, 3)) << std::endl;
  std::cout << "determinant2 " << det(submatrix(C, 3, 3, 4, 4)) << std::endl;
  std::cout << "determinant3 " << det(submatrix(C, 7, 7, 3, 3)) << std::endl;
  std::cout << "determinant4 " << det(submatrix(C, 10, 10, 4, 4)) << std::endl;
#endif
  
  return;
}

// Given a North-East-Down coordinate system at a point on a planet surface,
// left camera center, the x and y coordinates of where the ray from that
// center intersects the plane z = 0, and the same for the right camera,
// all in NED coordinates, find where the rays intersect, also in NED.
vw::Vector3 nedTri(vw::Vector3 const& cam1_ctr, vw::Vector3 const& cam2_ctr,
                   double x1, double y1, double x2, double y2) {

  // Find the normalized direction from camera to ground
  vw::Vector3 ground_pt1(x1, y1, 0.0);
  vw::Vector3 cam1_dir = ground_pt1 - cam1_ctr; cam1_dir /= norm_2(cam1_dir);
  vw::Vector3 ground_pt2(x2, y2, 0.0);
  vw::Vector3 cam2_dir = ground_pt2 - cam2_ctr; cam2_dir /= norm_2(cam2_dir);

  vw::Vector3 tri, err;
  tri = vw::stereo::triangulate_pair(cam1_dir, cam1_ctr, cam2_dir, cam2_ctr, err);
  
  return tri;
}

// Given a triangulated point in ECEF, create the local
// North-East-Down (NED) coordinate system centered at that
// point. Find the Jacobian of the nedTri() function, which will
// propagate uncertainties from the North-East horizontal plane
// through triangulation, with the result also being in NED.
// Bundle-adjusted cameras need no special treatment.
void triangulationJacobian(vw::cartography::Datum const& datum,
                           vw::Vector3 const& tri_nominal,
                           vw::camera::CameraModel const* cam1,
                           vw::camera::CameraModel const* cam2,
                           vw::Vector2 const& pix1,
                           vw::Vector2 const& pix2,
                           vw::Matrix<double> & J) {
  
  // The matrix to go from the NED coordinate system to ECEF at the
  // nominal triangulation point
  vw::Vector3 llh = datum.cartesian_to_geodetic(tri_nominal);
  vw::Matrix3x3 NedToEcef = datum.lonlat_to_ned_matrix(subvector(llh, 0, 2));
  vw::Matrix3x3 EcefToNed = inverse(NedToEcef);

  // Camera centers and directions in ECEF
  vw::Vector3 cam1_ctr = cam1->camera_center(pix1), cam1_dir = cam1->pixel_to_vector(pix1);
  vw::Vector3 cam2_ctr = cam2->camera_center(pix2), cam2_dir = cam2->pixel_to_vector(pix2);

  // Convert to NED
  vw::Vector3 cam1_ctr_ned = EcefToNed * (cam1_ctr - tri_nominal);
  vw::Vector3 cam1_dir_ned = EcefToNed * cam1_dir;
  vw::Vector3 cam2_ctr_ned = EcefToNed * (cam2_ctr - tri_nominal);
  vw::Vector3 cam2_dir_ned = EcefToNed * cam2_dir;

  // See where the rays intersect the local horizontal plane Find
  // alpha1 so that cam1_ctr_ned + alpha1 * cam1_dir_ned has 3rd
  // coordinate equal to zero
  double alpha1 = -cam1_ctr_ned.z() / cam1_dir_ned.z();
  double x1 = cam1_ctr_ned.x() + alpha1 * cam1_dir_ned.x();
  double y1 = cam1_ctr_ned.y() + alpha1 * cam1_dir_ned.y();
  double alpha2 = -cam2_ctr_ned.z() / cam2_dir_ned.z();
  double x2 = cam2_ctr_ned.x() + alpha2 * cam2_dir_ned.x();
  double y2 = cam2_ctr_ned.y() + alpha2 * cam2_dir_ned.y();
  
  // There are 4 input variables: x and y position in the horizontal
  // plane for the first camera, then for the second one. For each of
  // them must compute a centered difference. The output has 3
  // variables, the NED triangulation point.
  J.set_size(3, 4);
  J.set_zero();

  for (int coord = 0; coord < 4; coord++) {

    // Perturb one variable at a time
    double x1_plus = x1, x1_minus = x1, x2_plus = x2, x2_minus = x2;
    double y1_plus = y1, y1_minus = y1, y2_plus = y2, y2_minus = y2;
    if (coord == 0) {
      x1_minus += -deltaPosition;
      x1_plus  +=  deltaPosition;
    } else if (coord == 1) {
      y1_minus += -deltaPosition;
      y1_plus  +=  deltaPosition;
    } else if (coord == 2) {
      x2_minus += -deltaPosition;
      x2_plus  +=  deltaPosition;
    } else if (coord == 3) {
      y2_minus += -deltaPosition;
      y2_plus  +=  deltaPosition;
    }

    vw::Vector3 xyz_plus = nedTri(cam1_ctr_ned, cam2_ctr_ned,
                                  x1_plus, y1_plus, x2_plus, y2_plus);
    vw::Vector3 xyz_minus = nedTri(cam1_ctr_ned, cam2_ctr_ned,
                                   x1_minus, y1_minus, x2_minus, y2_minus);

    // Centered difference
    vw::Vector3 partial_deriv = (xyz_plus - xyz_minus) / (2.0 * deltaPosition);
    
    for (int row = 0; row < 3; row++) 
      J(row, coord) = partial_deriv[row];
  }

  return;
}

// Propagate the covariances. Return stddev. See the .h file for more info.
vw::Vector2 propagateCovariance(vw::Vector3 const& tri_nominal,
                                vw::cartography::Datum const& datum,
                                vw::camera::CameraModel const* cam1,
                                vw::camera::CameraModel const* cam2,
                                vw::Vector2 const& pix1,
                                vw::Vector2 const& pix2) {

  // Return right away if triangulation was not successful. The caller will set the result
  // to (0, 0, 0).
  if (tri_nominal == vw::Vector3(0, 0, 0) || tri_nominal != tri_nominal) 
    vw::vw_throw(vw::ArgumentErr() << "Could not compute the covariance.\n");

  vw::Matrix<double> J, C;

  vw::Vector2 const& stddev = asp::stereo_settings().horizontal_stddev; // alias
  vw::Vector2 variance;
  for (int s = 0; s < 2; s++)
    variance[s] = stddev[s] * stddev[s]; // square these to create variances  
  
  if (variance[0] > 0 && variance[1] > 0) {
    // The user set horizontal stddev
    triangulationJacobian(datum, tri_nominal, cam1, cam2, pix1, pix2, J);
    C = vw::math::identity_matrix(4);
    // The first two covariances are the left camera horizontal square stddev,
    // and last two are for the right camera.
    C(0, 0) = variance[0]; C(1, 1) = variance[0];
    C(2, 2) = variance[1]; C(3, 3) = variance[1];
  } else {
    // Will arrive here only for DG cameras and if the user did not
    // set --horizontal-stddev.  The Jacobian of the transform from
    // ephemeris and attitude to the triangulated point in NED
    // coordinates, multiplied by a scale factor.
    asp::scaledDGTriangulationJacobian(datum, cam1, cam2, pix1, pix2, J);
    
    // The input covariance, divided by the square of the above scale factor.
    asp::scaledDGSatelliteCovariance(cam1, cam2, pix1, pix2, C);
  }
  
  // Propagate the covariance
  // Per: https://en.wikipedia.org/wiki/Propagation_of_uncertainty#Non-linear_combinations
  vw::Matrix<double> JT = transpose(J);
  vw::Matrix<double> P = J * C * JT;

#if 0
  // Useful debug code
  std::cout << "NED covariance " << P << std::endl;
  vw::Vector<std::complex<double>> e;
  vw::math::eigen(P, e);
  std::cout << "Eigenvalues: " << e << std::endl;
#endif
  
  // Horizontal component is the square root of the determinant of the
  // upper-left 2x2 block (horizontal plane component), which is the
  // same as the square root of the product of eigenvalues of this
  // matrix.  Intuitively, the area of an ellipse is the product of
  // semi-axes, which is the product of eigenvalues. Then, a circle
  // with radius which is the square root of the product of semi-axes
  // has the same area.
  vw::Matrix2x2 H = submatrix(P, 0, 0, 2, 2);
  vw::Vector2 ans;
  ans[0] = sqrt(det(H));

  // Vertical component is the z variance
  ans[1] = P(2, 2);

  // Check for NaN. Then the caller will return the zero vector, which
  // signifies that the there is no valid data
  if (ans != ans) 
    vw::vw_throw(vw::ArgumentErr() << "Could not compute the covariance.\n");

  // Take the square root, so return the standard deviation
  return vw::Vector2(sqrt(ans[0]), sqrt(ans[1]));
}
  
} // end namespace asp
