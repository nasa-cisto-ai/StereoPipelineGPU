#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <asp/Camera/RPCModelGen.h>
#include <asp/Camera/RPCModel.h>
#include <vw/Math/Geometry.h>

using namespace vw;

namespace asp {

  void unpackCoeffs(Vector<double> const& C,
                    RPCModel::CoeffVec& lineNum, RPCModel::CoeffVec& lineDen,
                    RPCModel::CoeffVec& sampNum, RPCModel::CoeffVec& sampDen
                    ){

    // There are 20 polynomial coefficients for each of lineNum,
    // lineDen, sampNum, sampDen, as the RPC polynomials are of degree
    // 3 in x and y.  However, the 0 degree coeffs for both
    // denominators is 1, so only 78 coefficients vary. Extract the
    // variable coefficients from a vector.

    VW_ASSERT(C.size() == 78,
              ArgumentErr() << "Must have 78 coefficients.\n");

    int start = 0;
    // Line
    subvector(lineNum, 0, 20) = subvector(C, start, 20); start += 20;
    lineDen[0] = 1;
    subvector(lineDen, 1, 19) = subvector(C, start, 19); start += 19;
    // Sample
    subvector(sampNum, 0, 20) = subvector(C, start, 20); start += 20;
    sampDen[0] = 1;
    subvector(sampDen, 1, 19) = subvector(C, start, 19); start += 19;

    VW_ASSERT(start == 78, ArgumentErr() << "Book-keeping error.\n");

    return;
  }

  void packCoeffs( RPCModel::CoeffVec const& lineNum, RPCModel::CoeffVec const& lineDen,
                   RPCModel::CoeffVec const& sampNum, RPCModel::CoeffVec const& sampDen,
                   Vector<double> & C
                   ){

    // This function does the reverse of unpackCoeffs().

    C.set_size(78);
    int start = 0;
    subvector(C, start, 20) = subvector(lineNum, 0, 20); start += 20;
    subvector(C, start, 19) = subvector(lineDen, 1, 19); start += 19;
    subvector(C, start, 20) = subvector(sampNum, 0, 20); start += 20;
    subvector(C, start, 19) = subvector(sampDen, 1, 19); start += 19;

    return;
  }

  /// Print out a name followed by the vector of values
  void print_vec(std::string const& name, Vector<double> const& vals){
    std::cout.precision(16);
    std::cout << name << ",";
    int len = vals.size();
    for (int i = 0; i < len - 1; i++)
      std::cout << vals[i] << ",";
    if (len > 0)
      std::cout << vals[len-1];
    std::cout << std::endl;
  }

  /// Dump a vector to a text file, one value per row.
  void print_vec_to_file(std::string const& path, Vector<double> const& vals) {
    std::ofstream outFile(path.c_str());
    outFile.precision(16);
    int len = vals.size();
    for (int i = 0; i < len - 1; i++)
      outFile << vals[i] << std::endl;
    if (len > 0)
      outFile << vals[len-1];
    outFile.close();

  }

  void write_levmar_solver_results(std::string const& output_prefix, int status,
                                   Vector<double> const& initial_params,
                                   Vector<double> const& final_params,
                                   Vector<double> const& actual_observation,
                                   RpcSolveLMA const& lma_model) {

    // Compute initial and final numbers
    Vector<double> initial_projected = lma_model(initial_params);
    Vector<double> final_projected   = lma_model(final_params);
    Vector<double> initial_error     = lma_model.difference(initial_projected, actual_observation);
    Vector<double> final_error       = lma_model.difference(final_projected,   actual_observation);

    // Log the solver status
    VW_OUT(VerboseDebugMessage, "math") << "rpc_gen: levmar solver status = " << status << std::endl;
    VW_OUT(VerboseDebugMessage, "math") << "rpc_gen: levmar solver initial error norm_2 = " << norm_2(initial_error) << std::endl;
    VW_OUT(VerboseDebugMessage, "math") << "rpc_gen: levmar solver final   error norm_2 = " << norm_2(final_error  ) << std::endl;

    //// Dump the values to file
    //print_vec_to_file(output_prefix + "_initial_parameters.csv", initial_params);
    //print_vec_to_file(output_prefix + "_final_parameters.csv",   final_params);
    //print_vec_to_file(output_prefix + "_initial_projected.csv", initial_projected);
    //print_vec_to_file(output_prefix + "_final_projected.csv",   final_projected);
    //print_vec_to_file(output_prefix + "_initial_error.csv",     initial_error);
    //print_vec_to_file(output_prefix + "_final_error.csv",       final_error);

    //// Also add the results to the log
    //VW_OUT(VerboseDebugMessage, "math") << "LM: starting proj  " << initial_projected << std::endl;
    //VW_OUT(VerboseDebugMessage, "math") << "LM: final    proj  " << final_projected   << std::endl;
    //VW_OUT(VerboseDebugMessage, "math") << "LM: starting error " << initial_error     << std::endl;
    //VW_OUT(VerboseDebugMessage, "math") << "LM: final    error " << final_error       << std::endl;
  }

  /// Computes a system solution from a seed and returns the final error number.
  int find_solution_from_seed(RpcSolveLMA    const& lma_model,
                              Vector<double> const& seed_params,
                              Vector<double> const& actual_observations,
                              Vector<double>      & final_params,
                              double              & norm_error) {

    // Initialize a zero vector of RPC model coefficients
    int status;

    // Use the L-M solver to optimize the RPC model coefficient values.
    const double abs_tolerance  = 1e-24;
    const double rel_tolerance  = 1e-24;
    const int    max_iterations = 2000;
    final_params = math::levenberg_marquardt( lma_model, seed_params, actual_observations,
                                              status, abs_tolerance, rel_tolerance,
                                              max_iterations );

    if (status < 1) { // This means the solver failed to converge!
      VW_OUT(DebugMessage, "asp") << "rpc_gen: WARNING --> Levenberg-Marquardt solver status = " << status << std::endl;
    }

    // Otherwise the solver converged, return the final error number.
    Vector<double> final_projected = lma_model(final_params);
    Vector<double> final_error     = lma_model.difference(final_projected, actual_observations);
    norm_error = norm_2(final_error);
    return status;
  }

  void gen_rpc(// Inputs
               double penalty_weight,
               std::string    const& output_prefix,
               Vector<double> const& normalized_geodetics,
               Vector<double> const& normalized_pixels,
               Vector3 const& llh_scale,
               Vector3 const& llh_offset,
               Vector2 const& uv_scale,
               Vector2 const& uv_offset,
               // Outputs
               RPCModel::CoeffVec & line_num,
               RPCModel::CoeffVec & line_den,
               RPCModel::CoeffVec & samp_num,
               RPCModel::CoeffVec & samp_den){
  
    VW_ASSERT( penalty_weight >= 0, ArgumentErr()
               << "The RPC penalty weight must be non-negative.\n" );
  
    // The percentage of the error that the penalty weights should represent
    double penalty_weight_fraction = penalty_weight;
    // Fraction with no adjustment
    double native_penalty_fraction
      = (double)RpcSolveLMA::NUM_PENALTY_TERMS / (double)normalized_pixels.size();
    double penalty_adjustment      = penalty_weight_fraction / native_penalty_fraction;
    
    VW_OUT(DebugMessage, "math") << "rpc_gen: Computed penalty weight: "
                                 << penalty_adjustment<< std::endl;

    // Initialize a specialized least squares solver object and load the input data
    RpcSolveLMA lma_model (normalized_geodetics, normalized_pixels, penalty_adjustment);

    Vector<double> solution;
    double norm_error;

    // As initial guess we use the linear part of the RPC transform.
    int numPts = normalized_geodetics.size()/RPCModel::GEODETIC_COORD_SIZE;
    int numPts2 = (normalized_pixels.size() - asp::RpcSolveLMA::NUM_PENALTY_TERMS)
      / RPCModel::IMAGE_COORD_SIZE;
    
    if (numPts != numPts2) 
      vw_throw( ArgumentErr() << "Error in " << __FILE__
                << ". Number of inputs and outputs do not agree.\n");
    std::vector< Vector<double, RPCModel::GEODETIC_COORD_SIZE+1> > in(numPts), out(numPts);
    for (int p = 0; p < numPts; p++) {

      Vector<double, RPCModel::GEODETIC_COORD_SIZE+1> P;
      for (int q = 0; q < RPCModel::GEODETIC_COORD_SIZE; q++) 
        P[q] = normalized_geodetics[p*RPCModel::GEODETIC_COORD_SIZE + q]; // P[0], P[1], P[2]
      P[RPCModel::GEODETIC_COORD_SIZE] = 1; // P[3]
      in[p] = P;
      
      for (int q = 0; q < RPCModel::IMAGE_COORD_SIZE; q++) 
        P[q] = normalized_pixels[p*RPCModel::IMAGE_COORD_SIZE + q]; // P[0], P[1]
      P[RPCModel::IMAGE_COORD_SIZE]   = 0; // P[2]
      P[RPCModel::IMAGE_COORD_SIZE+1] = 1; // P[3]
      out[p] = P;
    }
    Matrix4x4 T = math::AffineFittingFunctorN<RPCModel::GEODETIC_COORD_SIZE>()(in, out);

    // Put this matrix in the format acceptable for the RPC solver
    for (int p = 0; p < int(line_num.size()); p++) {
      samp_num[p] = 0; samp_den[p] = 0; // first coordinate of output is sample
      line_num[p] = 0; line_den[p] = 0; // second coordinate of output is line
    }

    // The first coordinate of the output
    samp_num[0] = T(0, 3); // the d value, the translation, in a*x + b*y + c*z + d
    samp_num[1] = T(0, 0); samp_num[2] = T(0, 1); samp_num[3] = T(0, 2); // linear part, a, b, c
    
    // The second coordinate of the output
    line_num[0] = T(1, 3); // the d value, the translation, in a*x + b*y + c*z + d
    line_num[1] = T(1, 0); line_num[2] = T(1, 1); line_num[3] = T(1, 2); // linear part, a, b, c

    // The denominator is just 1 to start
    samp_den[0] = 1;
    line_den[0] = 1;
    
    // Initialize the model
    Vector<double> startGuess;
    startGuess.set_size(RPCModel::NUM_RPC_COEFFS);
    // for (size_t i = 0; i < startGuess.size(); i++) startGuess[i] = 0.0; // start with zero
    packCoeffs(line_num, line_den, samp_num, samp_den, startGuess);

    VW_OUT(DebugMessage, "asp") << "Initial guess for RPC coeffs: " << startGuess << std::endl;
    
    // Use the L-M solver to optimize the RPC model coefficient values.
    int status = find_solution_from_seed(lma_model, startGuess, normalized_pixels,
                                         solution, norm_error);
    VW_OUT(DebugMessage, "asp") << "Solved RPC coeffs: " << solution << std::endl;
    VW_OUT(DebugMessage, "asp") << "rpc_gen: norm_error = " << norm_error << std::endl;

    // Dump all the results to disk if the user passed in an output prefix.
    if (output_prefix != "")
      write_levmar_solver_results(output_prefix, status, startGuess,
                                  solution, normalized_pixels, lma_model);
  
    // If we ever want to improve our results further we should
    // experiment with multiple starting seeds!

    unpackCoeffs(solution, line_num, line_den, samp_num, samp_den);
  }
  
  
}
