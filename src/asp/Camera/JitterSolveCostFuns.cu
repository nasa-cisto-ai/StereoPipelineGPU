#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Cost functions used in solving for jitter. These need access to the camera models,
// so they are stored in the Camera folder.

#include <asp/Camera/JitterSolveCostFuns.h>
#include <asp/Core/CameraTransforms.h>
#include <asp/Core/SatSimBase.h>

#include <vw/Cartography/GeoReferenceUtils.h>

namespace asp {

// See the .h file for the documentation.
bool LsPixelReprojErr::operator()(double const * const * parameters, 
                                  double * residuals) const {

  try {
    // Make a copy of the model, as we will update quaternion and position values
    // that are being modified now. This may be expensive.
    UsgsAstroLsSensorModel cam = *m_ls_model;

    // Update the relevant quaternions in the local copy
    int shift = 0;
    for (int qi = m_begQuatIndex; qi < m_endQuatIndex; qi++) {
      for (int coord = 0; coord < NUM_QUAT_PARAMS; coord++) {
        cam.m_quaternions[NUM_QUAT_PARAMS * qi + coord]
          = parameters[qi + shift - m_begQuatIndex][coord];
      }
    }

    // Same for the positions. Note how we move forward in the parameters array,
    // as this is after the quaternions
    shift += (m_endQuatIndex - m_begQuatIndex);
    for (int pi = m_begPosIndex; pi < m_endPosIndex; pi++) {
      for (int coord = 0; coord < NUM_XYZ_PARAMS; coord++) {
        cam.m_positions[NUM_XYZ_PARAMS * pi + coord]
          = parameters[pi + shift - m_begPosIndex][coord];
      }
    }

    // Move forward in the array of parameters, then recover the triangulated point
    shift += (m_endPosIndex - m_begPosIndex);
    csm::EcefCoord P;
    P.x = parameters[shift][0];
    P.y = parameters[shift][1];
    P.z = parameters[shift][2];

    // Project in the camera with high precision. Do not use here
    // anything lower than 1e-8, as the linescan model will then
    // return junk.
    double desired_precision = asp::DEFAULT_CSM_DESIRED_PRECISISON;
    csm::ImageCoord imagePt = cam.groundToImage(P, desired_precision);

    // Convert to what ASP expects
    vw::Vector2 pix;
    asp::fromCsmPixel(pix, imagePt);

    residuals[0] = m_weight*(pix[0] - m_observation[0]);
    residuals[1] = m_weight*(pix[1] - m_observation[1]);
    
  } catch (std::exception const& e) {
    residuals[0] = g_big_pixel_value;
    residuals[1] = g_big_pixel_value;
    return true; // accept the solution anyway
  }

  return true;
}

// See the .h file for the documentation.
bool FramePixelReprojErr::operator()(double const * const * parameters, 
                                     double * residuals) const {

  try {
    // Make a copy of the model, as we will update position and quaternion
    // values that are being modified now. Use the same order as in
    // UsdAstroFrameSensorModel::m_currentParameterValue.
    UsgsAstroFrameSensorModel cam = *m_frame_model;

    // The latest position is in parameters[0].
    for (int coord = 0; coord < NUM_XYZ_PARAMS; coord++)
      cam.setParameterValue(coord, parameters[0][coord]);

    // The latest quaternion is in parameters[1]. Note how we below
    // move forward when invoking cam.setParameterValue().
    for (int coord = 0; coord < NUM_QUAT_PARAMS; coord++) 
      cam.setParameterValue(coord + NUM_XYZ_PARAMS, parameters[1][coord]);

    // The triangulation parameter is after the position and orientation
    csm::EcefCoord P;
    P.x = parameters[2][0];
    P.y = parameters[2][1];
    P.z = parameters[2][2];

    // Project in the camera with high precision. Do not use here
    // anything lower than 1e-8, as the linescan model will then
    // return junk.
    double desired_precision = asp::DEFAULT_CSM_DESIRED_PRECISISON;
    csm::ImageCoord imagePt = cam.groundToImage(P, desired_precision);

    // Convert to what ASP expects
    vw::Vector2 pix;
    asp::fromCsmPixel(pix, imagePt);

    residuals[0] = m_weight*(pix[0] - m_observation[0]);
    residuals[1] = m_weight*(pix[1] - m_observation[1]);
    
  } catch (std::exception const& e) {
    residuals[0] = g_big_pixel_value;
    residuals[1] = g_big_pixel_value;
    return true; // accept the solution anyway
  }

  return true;
}

// Constructor for weightedRollYawError. See the .h file for the documentation.
weightedRollYawError::weightedRollYawError
                  (std::vector<double>           const& positions, 
                   std::vector<double>           const& quaternions,
                   vw::cartography::GeoReference const& georef,
                   int cur_pos, double rollWeight, double yawWeight,
                   bool initial_camera_constraint): 
                   m_rollWeight(rollWeight), m_yawWeight(yawWeight), 
                   m_initial_camera_constraint(initial_camera_constraint) {

    int num_pos = positions.size()/NUM_XYZ_PARAMS;
    int num_quat = quaternions.size()/NUM_QUAT_PARAMS;
    if (num_pos != num_quat)
      vw::vw_throw(vw::ArgumentErr() 
        << "weightedRollYawError: Expecting the same number of positions and quaternions.\n");
    if (cur_pos < 0 || cur_pos >= num_pos)
      vw::vw_throw(vw::ArgumentErr() 
        << "weightedRollYawError: Expecting position index in range.\n");

    // Find the nearest neighbors of the current position
    int beg_pos = std::max(0, cur_pos - 1);
    int end_pos = std::min(num_pos - 1, cur_pos + 1);
    if (beg_pos >= end_pos)
      vw::vw_throw(vw::ArgumentErr() 
        << "weightedRollYawError: Expecting at least 2 camera positions.\n");

    // Find the segment along which the cameras are located, in projected coordinates
    // Here we mirror the logic from SatSim.cc
    int b = beg_pos * NUM_XYZ_PARAMS;
    int c = cur_pos * NUM_XYZ_PARAMS;
    int e = end_pos * NUM_XYZ_PARAMS;
    vw::Vector3 beg_pt(positions[b], positions[b+1], positions[b+2]);
    vw::Vector3 cur_pt(positions[c], positions[c+1], positions[c+2]);
    vw::Vector3 end_pt(positions[e], positions[e+1], positions[e+2]);

    // Orbital points before the current one, the current one, and after the
    // current one, in projected coordinates
    vw::Vector3 beg_proj = vw::cartography::ecefToProj(georef, beg_pt);
    vw::Vector3 cur_proj = vw::cartography::ecefToProj(georef, cur_pt);
    vw::Vector3 end_proj = vw::cartography::ecefToProj(georef, end_pt);
    
    // Find satellite along and across track directions in projected coordinates
    vw::Vector3 proj_along, proj_across;
    asp::calcProjAlongAcross(beg_proj, end_proj, proj_along, proj_across);

    // Find along and across in ECEF
    vw::Vector3 along, across;
    asp::calcEcefAlongAcross(georef, asp::satSimDelta(), 
                              proj_along, proj_across, cur_proj,
                              along, across); // outputs

    // Find the z vector as perpendicular to both along and across
    vw::Vector3 down = vw::math::cross_prod(along, across);
    down = down / norm_2(down);

    // Find the rotation matrix from satellite to world coordinates, and 90
    // degree in-camera rotation. It is assumed, as in sat_sim, that:
    // cam2world = sat2World * rollPitchYaw * rotXY.
    asp::assembleCam2WorldMatrix(along, across, down, m_sat2World);
    m_rotXY = asp::rotationXY();

    // Initial camera rotation matrix, before we optimize it
    m_initCam2World = asp::quaternionToMatrix(&quaternions[cur_pos*NUM_QUAT_PARAMS]);
}

  // See the .h file for the documentation.
  bool weightedRollYawError::operator()(double const * const * parameters, 
                                        double * residuals) const {

    // Convert to rotation matrix. Order of quaternion is x, y, z, w.  
    vw::Matrix3x3 cam2world = asp::quaternionToMatrix(parameters[0]);

    if (m_initial_camera_constraint) {
      // Find the new camera orientation relative to the initial camera, not
      // relative to the satellite along-track direction. Then find the roll and
      // yaw from it. This is experimental.
      vw::Matrix3x3 cam2cam =  vw::math::inverse(cam2world) * m_initCam2World;

      double roll, pitch, yaw;
      rollPitchYawFromRotationMatrix(cam2cam, roll, pitch, yaw);

      // Fix for roll / yaw being determined with +/- 180 degree ambiguity.
      roll  = roll  - 180.0 * round(roll  / 180.0);
      pitch = pitch - 180.0 * round(pitch / 180.0);
      yaw   = yaw   - 180.0 * round(yaw   / 180.0);

      // Roll, pitch, yaw in camera coordinates are pitch, roll, yaw in satellite
      // coordinates. So adjust below accordingly.
      // CERES is very tolerant if one of the weights used below is 0. So there is
      // no need to use a special cost function for such cases.
      residuals[0] = pitch * m_rollWeight; // per above, swap roll and pitch
      residuals[1] = yaw  * m_yawWeight;

      return true;
    }

    vw::Matrix3x3 rollPitchYaw  
      = vw::math::inverse(m_sat2World) * cam2world * vw::math::inverse(m_rotXY);

    double roll, pitch, yaw;
    rollPitchYawFromRotationMatrix(rollPitchYaw, roll, pitch, yaw);

    // Fix for roll / yaw being determined with +/- 180 degree ambiguity.
    roll = roll - 180.0 * round(roll / 180.0);
    pitch = pitch - 180.0 * round(pitch / 180.0);
    yaw  = yaw  - 180.0 * round(yaw  / 180.0);

    // CERES is very tolerant if one of the weights used below is 0. So there is
    // no need to use a special cost function for such cases.
    residuals[0] = roll * m_rollWeight;
    residuals[1] = yaw  * m_yawWeight;

    return true;
  }

} // end namespace asp
