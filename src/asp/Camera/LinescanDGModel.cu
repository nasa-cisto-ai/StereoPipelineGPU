#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Camera/RPC_XML.h>
#include <asp/Camera/LinescanDGModel.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Camera/CsmModel.h>
#include <asp/Camera/CsmUtils.h>
#include <asp/Camera/Covariance.h>

#include <usgscsm/UsgsAstroLsSensorModel.h>
#include <usgscsm/Utilities.h>

// TODO(oalexan1): Convert this to populating a CSM model. Remove all legacy
// code also remove the old linescan jitter code that inherits from this.

using namespace vw;

namespace asp {

// -----------------------------------------------------------------
// LinescanDGModel supporting functions

boost::posix_time::ptime parse_dg_time(std::string str) {
  try{
    return boost::posix_time::time_from_string(str);
  }catch(...){
    // This is a useful error, it tells the user an XML camera file is
    // trying to be interpreted as a DG camera file.
    vw::vw_throw(vw::ArgumentErr() << "Failed to parse time from string: " << str << ". "
                 << "If you are not using Digital Globe images, you may need to "
                 << "specify the session type, such as -t rpc, -t rpcmaprpc, -t aster, etc.\n");
  }
  return boost::posix_time::time_from_string(str); // Never reached!
}

vw::CamPtr load_dg_camera_model_from_xml(std::string const& path) {

  // Parse the Digital Globe XML file
  GeometricXML geo;
  AttitudeXML  att;
  EphemerisXML eph;
  ImageXML     img;
  RPCXML       rpc;

  try {
    read_xml(path, geo, att, eph, img, rpc);
  } catch (const std::exception& e){
    vw::vw_throw(vw::ArgumentErr() << "Invalid Digital Globe XML file: " << path << ". "
                 << "If you are not using Digital Globe images, you may "
                 << "need to specify the session type, such as -t rpc, "
                 << "-t rpcmaprpc, -t aster, etc.\n"
		 << e.what() << "\n");
  }

  // For WV, only Stereo1B and Basic1B products are supported. Users
  // often say wrong results are produced with other products.
  std::string sat_id = img.sat_id; 
  boost::algorithm::to_lower(sat_id);
  std::string image_descriptor = img.image_descriptor; 
  boost::algorithm::to_lower(image_descriptor); 
  if (sat_id.size() >= 2 && sat_id.substr(0, 2) == "wv" && 
      image_descriptor != "stereo1b" && image_descriptor != "basic1b") {
    vw::vw_throw(vw::ArgumentErr() << "For WorldView images, only Stereo1B and Basic1B products are supported.\n");
  } 

  if (stereo_settings().dg_use_csm) 
    vw_out() << "Using the CSM model with DigitalGlobe cameras.\n";

  // Get an estimate of the surface elevation from the corners specified in the file.
  // - Not every file has this information, in which case we will just use zero.
  double mean_ground_elevation = 0.0;
  vw::BBox3 bbox = rpc.get_lon_lat_height_box();
  if (!bbox.empty())
    mean_ground_elevation = (bbox.min()[2] + bbox.max()[2]) / 2.0;
  
  // Convert measurements in millimeters to pixels.
  geo.principal_distance /= geo.detector_pixel_pitch;
  geo.detector_origin    /= geo.detector_pixel_pitch;

  // Convert all time measurements to something that boost::date_time can read.
  boost::replace_all(eph.start_time,            "T", " ");
  boost::replace_all(img.tlc_start_time,        "T", " ");
  boost::replace_all(img.first_line_start_time, "T", " ");
  boost::replace_all(att.start_time,            "T", " ");

  // Convert UTC time measurements to line measurements. Ephemeris
  // start time will be our reference frame to calculate seconds against.
  SecondsFromRef convert(parse_dg_time(eph.start_time));

  // It is assumed that EPH and ATT are sampled at the same rate and time.
  VW_ASSERT(eph.satellite_position_vec.size() == att.satellite_quat_vec.size(),
            vw::MathErr() << "Ephemeris and attitude don't have the same number of samples.");
  VW_ASSERT(eph.start_time == att.start_time && eph.time_interval == att.time_interval,
            vw::MathErr() << "Ephemeris and attitude don't seem to use the same t0 or dt.");

  // Load up the time interpolation class. If the TLCList only has
  // one entry, then we have to manually drop in the slope and offset.
  if (img.tlc_vec.size() == 1) {
    double direction = 1;
    if (boost::to_lower_copy(img.scan_direction) != "forward") {
      direction = -1;
    }
    img.tlc_vec.push_back(std::make_pair(img.tlc_vec.front().first +
                                         img.avg_line_rate, direction));
  }

  // Build the TLCTimeInterpolation object and do a quick sanity check.
  vw::camera::TLCTimeInterpolation
    tlc_time_interpolation(img.tlc_vec, convert(parse_dg_time(img.tlc_start_time)));
  
  VW_ASSERT(fabs(convert(parse_dg_time(img.first_line_start_time)) -
  tlc_time_interpolation(0)) < fabs(1.0 / (10.0 * img.avg_line_rate)),
	     vw::MathErr()
	     << "First line time and output from TLC lookup table "
	     << "do not agree of the ephemeris time for the first line of the image. "
	     << "If your XML camera files are not from the WorldView satellites, "
	     << "you may try the switch -t rpc to use the RPC camera model.\n"
	     << "The first image line ephemeris time is: "
  	     << convert(parse_dg_time(img.first_line_start_time)) << ".\n"
	     << "The TLC look up table time is: " << tlc_time_interpolation(0) << ".\n"
	     << "Maximum allowed difference is 1/10 of avg line rate, which is: "
	     << fabs(1.0 / (10.0 * img.avg_line_rate))
	     << ".\n");

  double et0 = convert(parse_dg_time(eph.start_time));
  double at0 = convert(parse_dg_time(att.start_time));
  double edt = eph.time_interval;
  double adt = att.time_interval;

  if ((stereo_settings().enable_correct_velocity_aberration ||
       stereo_settings().enable_correct_atmospheric_refraction) &&
      stereo_settings().dg_use_csm)
    vw::vw_throw(vw::ArgumentErr() << "Cannot correct velocity aberration or "
                 << "atmospheric refraction with the CSM model.\n");
  
  vw::Quat sensor_rotation = vw::math::euler_xyz_to_quaternion
    (vw::Vector3(0,0,geo.detector_rotation - M_PI/2)); // explained earlier
  vw::Quat sensor_to_body = geo.camera_attitude * sensor_rotation;
  vw::Vector2 final_detector_origin
    = subvector(inverse(sensor_rotation).rotate(vw::Vector3(geo.detector_origin[0],
							      geo.detector_origin[1], 0)), 0, 2);

  // We will create one camera model in regular use, and 14 more of
  // them with slight perturbations if needed for error propagation
  // (covariance computation). This approach results in avoiding
  // writing a lot of new code which would be in some places similar
  // and in others different than existing one. See Covariance.h for
  // more details.
  vw::CamPtr nominal_cam;
  std::vector<vw::CamPtr> perturbed_cams;
  int num_cams = 1;
  if (asp::stereo_settings().propagate_errors)
    num_cams = numCamsForCovariance();

  for (int cam_it = 0; cam_it < num_cams; cam_it++) {
    vw::Vector<double, 3> dp = asp::positionDelta(cam_it);
    vw::Vector<double, 4> dq = asp::quatDelta(cam_it);

    // Convert ephemeris from satellite to camera position. Change
    // attitude to be the rotation from camera frame to world
    // frame. We also add an additional 90 degree rotation to the
    // camera frame so X is the horizontal direction to the picture
    // and +Y points down the image (in the direction of flight). Must
    // apply any perturbations when still in satellite coordinates, to
    // be consistent with input covariances.
    std::vector<vw::Vector3> camera_position_vec(eph.satellite_position_vec.size());
    std::vector<vw::Quat>    camera_quat_vec(att.satellite_quat_vec.size());
    for (size_t i = 0; i < eph.satellite_position_vec.size(); i++) {
      Vector<double, 3> p = eph.satellite_position_vec[i] + dp; // add the perturbation
      Vector<double, 4> q = att.satellite_quat_vec[i];
      // The dq perturbations are chosen under the assumption that q is normalized
      double len_q = norm_2(q);
      if (len_q > 0 && asp::stereo_settings().propagate_errors) 
        q = q / len_q; // Normalization is not needed without covariance logic
      q = q + dq;
      vw::Quat qt(q[3], q[0], q[1], q[2]); // Note the swapping, the order is now w, x, y, z.
      camera_position_vec[i] = p + qt.rotate(geo.perspective_center);
      camera_quat_vec[i] = qt * sensor_to_body;
    }

    vw::CamPtr cam_ptr
      (new DGCameraModel(vw::camera::PiecewiseAPositionInterpolation(camera_position_vec,
                                                                     eph.velocity_vec, et0, edt),
                         vw::camera::LinearPiecewisePositionInterpolation(eph.velocity_vec,
                                                                          et0, edt),
                         vw::camera::SLERPPoseInterpolation(camera_quat_vec, at0, adt),
                         tlc_time_interpolation, img.image_size, final_detector_origin,
                         geo.principal_distance, mean_ground_elevation,
                         stereo_settings().enable_correct_velocity_aberration,
                         stereo_settings().enable_correct_atmospheric_refraction));

    if (cam_it == 0) 
      nominal_cam = cam_ptr;
    else
      perturbed_cams.push_back(cam_ptr);
  }

  DGCameraModel * cam = (DGCameraModel*)nominal_cam.get();
  // Store the starting time and spacing for the satellite, in case later some resampling
  // happens to the cameras when those would use a different spacing.
  cam->m_satellite_pos_t0 = et0;
  cam->m_satellite_pos_dt = edt;
  cam->m_satellite_quat_t0 = at0;
  cam->m_satellite_quat_dt = adt;
  if (asp::stereo_settings().propagate_errors) {
    cam->m_perturbed_cams = perturbed_cams; 
    cam->m_satellite_pos_cov = eph.satellite_pos_cov;
    cam->m_satellite_quat_cov = att.satellite_quat_cov;
  }
  
  return nominal_cam;
} // End function load_dg_camera_model()

// Constructor
DGCameraModel::DGCameraModel
(vw::camera::PiecewiseAPositionInterpolation      const& position,
 vw::camera::LinearPiecewisePositionInterpolation const& velocity,
 vw::camera::SLERPPoseInterpolation               const& pose,
 vw::camera::TLCTimeInterpolation                 const& time,
 vw::Vector2i                                     const& image_size, 
 vw::Vector2                                      const& detector_origin,
 double                                           const  focal_length,
 double                                           const  mean_ground_elevation,
 bool                                                    correct_velocity,
 bool                                                    correct_atmosphere):
  DGCameraModelBase(position, velocity, pose, time, image_size, detector_origin, focal_length,
                    mean_ground_elevation, correct_velocity, correct_atmosphere) {
  
  // It is convenient to have the CSM model exist even if it is not used.
  // The cam_test.cc and jitter_solve.cc tools uses this assumption.
  // Soon the other implementation will go away and this will be the default.
  populateCsmModel();
}
  
// This is a lengthy function that does many initializations  
void DGCameraModel::populateCsmModel() {

  // Using a desired precision of 1e-8 will result in about this much
  // agreement between image to ground and back. Pushing this to
  // something lower will result in the CSM ground-to-image
  // computation failing due to numerical precision issues, which can
  // be traced to the DG camera using a focal length (in pixels) of
  // 2,002,252.25.
  m_csm_model.reset(new CsmModel);

  // this sensor is used for Earth only
  vw::cartography::Datum datum = vw::cartography::Datum("WGS84"); 
  m_csm_model->m_desired_precision = asp::DEFAULT_CSM_DESIRED_PRECISISON;
  m_csm_model->m_semi_major_axis = datum.semi_major_axis(); // WGS84
  m_csm_model->m_semi_minor_axis = datum.semi_minor_axis(); // WGS84
    
  // Create a linescan model as a smart pointer, and do smart pointer
  // casting Follow the CSM API. The type of m_gm_model is
  // csm::RasterGM, which is a base class. UsgsAstroLsSensorModel is
  // derived from it. A smart pointer to m_gm_model is held by
  // m_csm_model.
  m_csm_model->m_gm_model.reset(new UsgsAstroLsSensorModel);
  m_ls_model = boost::dynamic_pointer_cast<UsgsAstroLsSensorModel>
    (m_csm_model->m_gm_model);
  if (m_ls_model == NULL)
    vw::vw_throw(vw::ArgumentErr() << "Invalid initialization of the linescan model.\n");
    
  // This performs many initializations apart from the above. Note that this is
  // not a boost::shared_ptr reset, it is UsgsAstroLsSensorModel reset.
  m_ls_model->reset();
    
  m_ls_model->m_nSamples = m_image_size[0]; 
  m_ls_model->m_nLines   = m_image_size[1];

  double f = m_focal_length;

  m_ls_model->m_platformFlag = 1; // For order 8 Lagrange interpolation
  m_ls_model->m_maxElevation =  10000.0; //  10 km
  m_ls_model->m_minElevation = -10000.0; // -10 km
  m_ls_model->m_focalLength  =  f;
  m_ls_model->m_zDirection   = 1.0;
  m_ls_model->m_halfSwath    = 1.0;
  m_ls_model->m_sensorIdentifier = "DigitalGlobeLinescan";
  m_ls_model->m_majorAxis = m_csm_model->m_semi_major_axis;
  m_ls_model->m_minorAxis = m_csm_model->m_semi_minor_axis;

  // The choices below are because of how DigitalGlobe's
  // get_local_pixel_vector() interacts with the
  // UsgsAstroLsSensorModel function
  // computeDistortedFocalPlaneCoordinates(). For Pleiades
  // get_local_pixel_vector() is computed differently, and hence
  // different choices here as well. Also keep in mind that a CSM
  // pixel has extra 0.5 added to it.
  m_ls_model->m_iTransL[0]             = 0.0;  
  m_ls_model->m_iTransL[1]             = 0.0;
  m_ls_model->m_iTransL[2]             = 1.0;
  m_ls_model->m_iTransS[0]             = 0.0;
  m_ls_model->m_iTransS[1]             = 1.0;
  m_ls_model->m_iTransS[2]             = 0.0;
  m_ls_model->m_detectorLineOrigin     = 0.0;
  m_ls_model->m_detectorSampleOrigin   = 0.0;
  m_ls_model->m_detectorLineSumming    = 1.0;
  // TODO(oalexan1): Consider subtracting 0.5 below, and then adding
  // only 0.5 to tlc[i].first, further down. Looks to produce similar
  // but not quite the same result, but there was no time for a lot of
  // testing doing it that way.
  m_ls_model->m_startingDetectorLine   = m_detector_origin[1];
  m_ls_model->m_detectorSampleSumming  = 1.0;
  m_ls_model->m_startingDetectorSample = (m_detector_origin[0] - 0.5);

  // Time
  auto const& tlc = m_time_func.m_tlc;
  double time_offset = m_time_func.m_time_offset;
  if (tlc.size() < 2) 
    vw::vw_throw(vw::ArgumentErr()
                 << "Expecting at least two line and time sample pairs.\n");
  m_ls_model->m_intTimeLines.clear(); // not needed, but best for clarity
  m_ls_model->m_intTimeStartTimes.clear();
  m_ls_model->m_intTimes.clear();
  for (size_t i = 0; i < tlc.size(); i++) {
    m_ls_model->m_intTimeLines.push_back(tlc[i].first + 1.0); // line
    m_ls_model->m_intTimeStartTimes.push_back(tlc[i].second + time_offset); // time
    // Slope
    if (i + 1 < tlc.size()) {
      // Compute the slope between this time instance and the next
      double slope = (tlc[i+1].second - tlc[i].second) / (tlc[i+1].first - tlc[i].first);
      m_ls_model->m_intTimes.push_back(slope);
    } else{
      // Cannot have a slope for the last value, as there's no next one to use,
      // but for consistency, borrow the last slope. This is consistent
      // with lines out of range using the slopes closest to them.
      double slope = m_ls_model->m_intTimes.back();
      m_ls_model->m_intTimes.push_back(slope);
    }
  }

  // Compute positions and velocities. DigitalGlobe uses linear
  // interpolation based on tabulated positions and velocities to find
  // any given position, via the class
  // PiecewiseAPositionInterpolation. But UsgsAstroLsSensorModel
  // expects Lagrange interpolation of positions only, and makes no
  // use of velocities for that. For backward compatibility, provide
  // UsgsAstroLsSensorModel with more position samples by a given
  // factor, with each sample obtained with
  // PiecewiseAPositionInterpolation, so it makes use of both
  // positions and velocities.
  // TODO(oalexan1): Are positions sampled finely enough?
  int factor = 1;
  double old_t0 = m_position_func.get_t0();
  double old_dt = m_position_func.get_dt();
  double old_tend = m_position_func.get_tend();
  int num_old_pos = round((old_tend - old_t0)/old_dt) + 1;
  int num_new_pos = factor * (num_old_pos - 1) + 1; // care here, to not go out of bounds
  m_ls_model->m_numPositions = 3 * num_new_pos; // concatenate all coordinates
  m_ls_model->m_t0Ephem = old_t0; // same starting position
  m_ls_model->m_dtEphem = old_dt / factor; // finer sampling
  m_ls_model->m_positions.resize(m_ls_model->m_numPositions);
  m_ls_model->m_velocities.resize(m_ls_model->m_numPositions);
  for (int pos_it = 0; pos_it < num_new_pos; pos_it++) {
    // The largest value of t will be no more than old_tend, within numerical precision
    double t = m_ls_model->m_t0Ephem + pos_it * m_ls_model->m_dtEphem;
    vw::Vector3 P = m_position_func(t);
    vw::Vector3 V = m_velocity_func(t);
    for (int coord = 0; coord < 3; coord++) {
      m_ls_model->m_positions [3*pos_it + coord] = P[coord];
      m_ls_model->m_velocities[3*pos_it + coord] = V[coord];
    }
  }

  // Quaternions
  // TODO(oalexan1): Are the quaternions sampled finely enough?
  // Since DG has a lot of them, we assume there's no need for more.
  // ASP's old approach used linear quaternion interpolation,
  // but CSM uses Lagrange interpolation.
  int num_quat = m_pose_func.m_pose_samples.size();
  m_ls_model->m_numQuaternions = 4 * num_quat; // concatenate all coordinates
  m_ls_model->m_t0Quat = m_pose_func.m_t0; // quaternion t0
  m_ls_model->m_dtQuat = m_pose_func.m_dt; // quaternion dt
  m_ls_model->m_quaternions.resize(m_ls_model->m_numQuaternions);

  for (int pos_it = 0; pos_it < m_ls_model->m_numQuaternions / 4; pos_it++) {
    double t = m_ls_model->m_t0Quat + pos_it * m_ls_model->m_dtQuat;
    vw::Quat q = m_pose_func.m_pose_samples[pos_it];
    // ASP stores the quaternions as (w, x, y, z). CSM wants them as
    // x, y, z, w.
    int coord = 0;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.x(); coord++;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.y(); coord++;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.z(); coord++;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.w(); coord++;
  }

  // Re-creating the model from the state forces some operations to
  // take place which are inaccessible otherwise.
  std::string modelState = m_ls_model->getModelState();
  m_ls_model->replaceModelState(modelState);
}

// Re-implement base class functions
  
double DGCameraModel::get_time_at_line(double line) const {
  
  if (stereo_settings().dg_use_csm) {
    csm::ImageCoord csm_pix;
    vw::Vector2 pix(0, line);
    asp::toCsmPixel(pix, csm_pix);
    return m_ls_model->getImageTime(csm_pix);
  }
  
  return m_time_func(line);
}

vw::Vector3 DGCameraModel::get_camera_center_at_time(double time) const {
  if (stereo_settings().dg_use_csm) {
    csm::EcefCoord ecef = m_ls_model->getSensorPosition(time);
    return vw::Vector3(ecef.x, ecef.y, ecef.z);
  }
  
  return m_position_func(time);
}

vw::Vector3 DGCameraModel::get_camera_velocity_at_time(double time) const {
  if (stereo_settings().dg_use_csm) {
    csm::EcefVector ecef = m_ls_model->getSensorVelocity(time);
    return vw::Vector3(ecef.x, ecef.y, ecef.z);
  }
  
  return m_velocity_func(time);
}

// Function to interpolate quaternions with the CSM model. This is used
// for CSM model validation but not in production.
// TODO(oalexan1): Move this to a new CsmModelUtils.cc file and call it from here.
void DGCameraModel::getQuaternions(const double& time, double q[4]) const {

  if (!stereo_settings().dg_use_csm)
    vw::vw_throw(vw::ArgumentErr()
                 << "getQuaternions: It was expected that the CSM model was used.\n");
    
  int nOrder = 8;
  if (m_ls_model->m_platformFlag == 0)
    nOrder = 4;
  int nOrderQuat = nOrder;
  if (m_ls_model->m_numQuaternions/4 < 6 && nOrder == 8)
    nOrderQuat = 4;

  lagrangeInterp(m_ls_model->m_numQuaternions / 4,
                 &m_ls_model->m_quaternions[0],
                 m_ls_model->m_t0Quat, m_ls_model->m_dtQuat,
                 time, 4, nOrderQuat, q);
}

// Interpolate the satellite position covariance at given pixel
void DGCameraModel::interpSatellitePosCov(vw::Vector2 const& pix,
                                          double p_cov[SAT_POS_COV_SIZE]) const {
  
  if (!stereo_settings().dg_use_csm)
    vw::vw_throw(vw::ArgumentErr()
                 << "interpSatellitePosCov: It was expected that the CSM model was used.\n");

  double time = get_time_at_line(pix.y());

  int numCov = m_satellite_pos_cov.size() / SAT_POS_COV_SIZE;

  int nOrder = 8;
  if (m_ls_model->m_platformFlag == 0)
    nOrder = 4;

  lagrangeInterp(numCov, &m_satellite_pos_cov[0], m_satellite_pos_t0, m_satellite_pos_dt,
                 time, SAT_POS_COV_SIZE, nOrder, p_cov);
  //asp::nearestNeibInterp(numCov, &m_satellite_pos_cov[0], m_satellite_pos_t0, 
  //                       m_satellite_pos_dt, time, SAT_POS_COV_SIZE, p_cov);
}

// Interpolate the satellite quaternion covariance at given pixel
void DGCameraModel::interpSatelliteQuatCov(vw::Vector2 const& pix,
                                           double q_cov[SAT_QUAT_COV_SIZE]) const {

  if (!stereo_settings().dg_use_csm)
    vw::vw_throw(vw::ArgumentErr()
                 << "interpSatelliteQuatCov: It was expected that the CSM model was used.\n");

  double time = get_time_at_line(pix.y());
  int numCov = m_satellite_quat_cov.size() / SAT_QUAT_COV_SIZE;
  
  int nOrder = 8;
  if (m_ls_model->m_platformFlag == 0)
    nOrder = 4;
  int nOrderQuat = nOrder;
  if (numCov < 6 && nOrder == 8)
    nOrderQuat = 4;

  //lagrangeInterp(numCov, &m_satellite_quat_cov[0], m_satellite_quat_t0, m_satellite_quat_dt,
  //               time, SAT_QUAT_COV_SIZE, nOrderQuat, q_cov);
  asp::nearestNeibInterp(numCov, &m_satellite_quat_cov[0], 
                    m_satellite_quat_t0, m_satellite_quat_dt,
                    time, SAT_QUAT_COV_SIZE, q_cov);
}

vw::Quat DGCameraModel::get_camera_pose_at_time(double time) const {
  if (stereo_settings().dg_use_csm) {
    double q[4];
    getQuaternions(time, q);
    return vw::Quat(q[3], q[0], q[1], q[2]); // go from (x, y, z, w) to (w, x, y, z)
  }
  
  return m_pose_func(time);
}
  
// Gives a pointing vector in the world coordinates.
vw::Vector3 DGCameraModel::pixel_to_vector(vw::Vector2 const& pix) const {

  if (stereo_settings().dg_use_csm) {
    csm::ImageCoord csm_pix;
    asp::toCsmPixel(pix, csm_pix);
    csm::EcefLocus locus = m_ls_model->imageToRemoteImagingLocus(csm_pix);
    return vw::Vector3(locus.direction.x, locus.direction.y, locus.direction.z);
  }
  
  return vw::camera::LinescanModel::pixel_to_vector(pix);
}

// As pixel_to_vector, but in the local camera frame.
vw::Vector3 DGCameraModel::get_local_pixel_vector(vw::Vector2 const& pix) const {
  if (stereo_settings().dg_use_csm)
    vw::vw_throw(vw::ArgumentErr()
                 << "get_local_pixel_vector(): Cannot be called in CSM mode.\n");
  
  vw::Vector3 local_vec(pix[0] + m_detector_origin[0], m_detector_origin[1], m_focal_length);
  return normalize(local_vec);
}

// See the .h file for the documentation.
double DGCameraModel::errorFunc(double y, vw::Vector3 const& point) const {

  double t = get_time_at_line(y);
  vw::Quat q = get_camera_pose_at_time(t);
  vw::Vector3 pt = inverse(q).rotate(point - get_camera_center_at_time(t));

  return pt.y() / pt.z() - m_detector_origin[1] / m_focal_length;
}
  
// Point to pixel with no initial guess
vw::Vector2 DGCameraModel::point_to_pixel(vw::Vector3 const& point) const {
  if (stereo_settings().dg_use_csm) {

    csm::EcefCoord ecef(point[0], point[1], point[2]);
    
    // Do not show warnings, it becomes too verbose
    double achievedPrecision = -1.0;
    csm::WarningList warnings;
    csm::WarningList * warnings_ptr = NULL;
    bool show_warnings = false;
    // Do not use here a desired precision than than 1e-8, as
    // then CSM can return junk.
    csm::ImageCoord csm_pix
      = m_ls_model->groundToImage(ecef,
                                  m_csm_model->m_desired_precision,
                                  &achievedPrecision, warnings_ptr);
    
    vw::Vector2 asp_pix;
    asp::fromCsmPixel(asp_pix, csm_pix);
    
#if 0
    // This logic is from UsgsAstroLsSensorModel, with fixes to make
    // it more robust for DG cameras. Not used yet.
    
    // TODO(oalexan1): Use this logic
    // with non-CSM cameras.  Should be much faster than the current
    // approach of solving a minimization problem.  Do not set
    // desired_precision to less than 1e-8 as then the algorithm will
    // produce junk due to numerical precision issues with the large
    // DG focal length.

    // For non-CSM logic, need to start iterating from sensor
    // midpoint.  asp_pix[1] = m_image_size.y()/2;
    
    double L0 = 0.0; // Line increment
    double lineErr0 = errorFunc(L0 + asp_pix[1], point);
    double L1 = 0.1;
    double lineErr1 = errorFunc(L1 + asp_pix[1], point);
    
    for (int count = 0; count < 15; count++) {
      
      if (lineErr1 == lineErr0)
        break; // avoid division by 0
      
      // Secant method update
      // https://en.wikipedia.org/wiki/Secant_method
      double increment = lineErr1 * (L1 - L0) / (lineErr1 - lineErr0);
      double L2 = L1 - increment;
      double lineErr2 = errorFunc(L2 + asp_pix[1], point);
      
      // Update for the next step
      L0 = L1; lineErr0 = lineErr1;
      L1 = L2; lineErr1 = lineErr2;
      
      // If the solution changes by less than this, we achieved the desired line precision
      if (increment < m_csm_model->m_desired_precision) 
        break;
    }
    
    asp_pix[1] += L1;
    
    // Solve for sample location now that we know the correct line
    double t = get_time_at_line(asp_pix[1]);
    vw::Quat q = get_camera_pose_at_time(t);
    vw::Vector3 pt = inverse(q).rotate(point - get_camera_center_at_time(t));
    pt *= m_focal_length / pt.z();
    asp_pix = vw::Vector2(pt.x() - m_detector_origin[0], asp_pix[1]);
#endif

    return asp_pix;
  }
  
  // Non-CSM version
  return vw::camera::LinescanModel::point_to_pixel(point);

}
  
// TODO(oalexan1): Wipe this and use the logic above, after much testing.  
vw::Vector2 DGCameraModel::point_to_pixel(vw::Vector3 const& point, double starty) const {

  if (stereo_settings().dg_use_csm)
    vw::vw_throw(vw::ArgumentErr()
                 << "point_to_pixel(point, starty): Cannot be called in CSM mode.\n");
    
  // Use the uncorrected function to get a fast but good starting seed.
  vw::camera::CameraGenericLMA model(this, point);
  int status = -1;
  vw::Vector2 start = point_to_pixel_uncorrected(point, starty);
  
  // Run the solver
  vw::Vector3 objective(0, 0, 0);
  const double ABS_TOL = 1e-16;
  const double REL_TOL = 1e-16;
  const int    MAX_ITERATIONS = 1e+5;
  vw::Vector2 solution = vw::math::levenberg_marquardtFixed<vw::camera::CameraGenericLMA, 2,3>
    (model, start, objective, status,
     ABS_TOL, REL_TOL, MAX_ITERATIONS);
  VW_ASSERT(status > 0,
            vw::camera::PointToPixelErr() << "Unable to project point into LinescanDG model.");
  return solution;
}
  
// Camera pose
vw::Quaternion<double> DGCameraModel::camera_pose(vw::Vector2 const& pix) const {

  if (stereo_settings().dg_use_csm) {
    vw_throw(vw::NoImplErr() << "camera_pose() is not implemented with CSM.");
    return vw::Quaternion<double>();
  }
  
  return vw::camera::LinescanModel::camera_pose(pix);
}

// Gives the camera position in world coordinates.
vw::Vector3 DGCameraModel::camera_center(vw::Vector2 const& pix) const {

  if (stereo_settings().dg_use_csm) {
    csm::ImageCoord csm_pix;
    asp::toCsmPixel(pix, csm_pix);
    
    double time = m_ls_model->getImageTime(csm_pix);
    csm::EcefCoord ecef = m_ls_model->getSensorPosition(time);
    
    return vw::Vector3(ecef.x, ecef.y, ecef.z);
  }
  
  return vw::camera::LinescanModel::camera_center(pix);
}
    
} // end namespace asp

