#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Camera/CameraSolve.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Camera/PeruSatXML.h>
#include <asp/Camera/LinescanPeruSatModel.h>

namespace asp {

// While static variables are not thread-safe, this will be changed only once,
// the first time a PeruSatCameraModel is loaded, and model loading is a serial
// process. 
static bool perusat_correction_note_printed = false;

using vw::Vector3;
using vw::Matrix3x3;

// TODO: Port these changes to the base class

vw::Vector2 PeruSatCameraModel::point_to_pixel(Vector3 const& point, double starty) const {

  // Use the generic solver to find the pixel 
  // - This method will be slower but works for more complicated geometries
  vw::camera::CameraGenericLMA model(this, point);
  int status;
  vw::Vector2 start = m_image_size / 2.0; // Use the center as the initial guess
  if (starty >= 0) // If the user provided a line number guess, use it.
    start[1] = starty;

  // Solver constants
  const double ABS_TOL = 1e-16;
  const double REL_TOL = 1e-16;
  const int    MAX_ITERATIONS = 1e+5;
  const double MAX_ERROR = 0.01;

  Vector3 objective(0, 0, 0);
  vw::Vector2 solution = vw::math::levenberg_marquardtFixed<vw::camera::CameraGenericLMA, 2,3>
    (model, start, objective, status, ABS_TOL, REL_TOL, MAX_ITERATIONS);

  double  error = norm_2(model(solution));

  VW_ASSERT( (status > 0) && (error < MAX_ERROR),
             vw::camera::PointToPixelErr()
             << "Unable to project point into LinescanPeruSat model" );

  return solution;
}

void PeruSatCameraModel::check_time(double time, std::string const& location) const {
  if ((time < m_min_time) || (time > m_max_time))
    vw::vw_throw(vw::ArgumentErr() << "PeruSatCameraModel::"<<location
                 << ": Requested time "<<time<<" is out of bounds ("
                 << m_min_time << " <-> "<<m_max_time<<")\n");
}

vw::Vector3 PeruSatCameraModel::get_camera_center_at_time(double time) const {
  check_time(time, "get_camera_center_at_time");
  return m_position_func(time);
}
vw::Vector3 PeruSatCameraModel::get_camera_velocity_at_time(double time) const { 
  check_time(time, "get_camera_velocity_at_time");
  return m_velocity_func(time); 
}
vw::Quat PeruSatCameraModel::get_camera_pose_at_time(double time) const {
  check_time(time, "get_camera_pose_at_time");
 return m_pose_func(time); 
}

double PeruSatCameraModel::get_time_at_line(double line) const {
  // Allow finding the time at any line, even negative ones.  Here a
  // simple slope-intercept formula is used rather than a table so one
  // cannot run out of bounds.
  return m_time_func(line); 
}

Vector3 PeruSatCameraModel::get_local_pixel_vector(vw::Vector2 const& pix) const {

  // According to Modelo%20Orbital%20PeruSAT-1.pdf:
  
  // psi_x = tan_psi_x[0] * (col - col_ref) + tan_psi_x[1]
  // psi_y = tan_psi_y[0] * (col - col_ref) + tan_psi_y[1]

  // where those coefficients are given in LINE_OF_SIGHT_TANPSIX
  // and LINE_OF_SIGHT_TANPSIY.

  // The doc says col_ref is 1, so for us it will be 0 since our
  // columns start form 0.

  double col     = pix[0];
  double psi_x   = m_tan_psi_x[0] * col + m_tan_psi_x[1];
  double psi_y   = m_tan_psi_y[0] * col + m_tan_psi_y[1];
  Vector3 result = Vector3(tan(psi_y), -tan(psi_x), 1.0);

  // Make the direction have unit length
  result = normalize(result);
  
  // Go from sensor coordinates to satellite body coordinates.
  // The instrument biases were not documented at all and took a while
  // to figure out.
  result = m_inverse_instrument_biases.rotate(result);

  return result;
}

boost::shared_ptr<PeruSatCameraModel> load_perusat_camera_model_from_xml(std::string const& path){

  vw_out(vw::DebugMessage,"asp") << "Loading PeruSat camera file: " << path << std::endl;
  // Parse the PeruSat XML file
  PeruSatXML xml_reader;
  xml_reader.read_xml(path);

  // Get all the initial functors
  vw::camera::LinearTimeInterpolation
    time_func      = xml_reader.setup_time_func();
  vw::camera::LagrangianInterpolation
    position_func  = xml_reader.setup_position_func(time_func);
  vw::camera::LagrangianInterpolation
    velocity_func  = xml_reader.setup_velocity_func(time_func);
  vw::camera::SLERPPoseInterpolation
    pose_func      = xml_reader.setup_pose_func(time_func);

  // Find the range of times for which we can solve for position and pose
  double min_position_time = position_func.get_t0();
  double max_position_time = position_func.get_tend();
  double min_velocity_time = velocity_func.get_t0();
  double max_velocity_time = velocity_func.get_tend();
  double min_pose_time = pose_func.get_t0();
  double max_pose_time = pose_func.get_tend();
  double min_time = std::max(min_position_time, std::max(min_velocity_time, min_pose_time));
  double max_time = std::min(max_position_time, std::min(max_velocity_time, max_pose_time));

  // See note on this below
  bool correct_velocity_aberration = false;
  bool correct_atmospheric_refraction = false;
  
  // Create the model. This can throw an exception.
  boost::shared_ptr<PeruSatCameraModel> cam
    (new PeruSatCameraModel(position_func, velocity_func, 
                            pose_func, time_func, 
                            xml_reader.m_tan_psi_x,
                            xml_reader.m_tan_psi_y,
                            xml_reader.m_instrument_biases,
                            xml_reader.m_image_size,
                            min_time, max_time,
                            correct_velocity_aberration,
                            correct_atmospheric_refraction));

  // Print this note only if PeruSat model loading was successful, as
  // sometimes this camera loading function is invoked when querying
  // an unknown XML model and we may not end up using this session if
  // loading fails.
  if (!perusat_correction_note_printed) {
    vw::vw_out() << "Not using atmospheric and velocity aberration correction "
             << "with PeruSat cameras to maintain closer agreement with "
             << "the RPC approximation to this model.\n";
    perusat_correction_note_printed = true;
  }
  
  return cam;
} // End function load_perusat_camera_model()

} // end namespace asp

