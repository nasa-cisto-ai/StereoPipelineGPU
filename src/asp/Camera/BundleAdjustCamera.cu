#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file BundleAdjustCamera.cc
///

// TODO(oalexan1): Move most of BundleAdjustCamera.h code to here, and put it
// all in the asp namespace. 

#include <asp/Camera/BundleAdjustCamera.h>
#include <asp/Core/IpMatchingAlgs.h>         // Lightweight header

#include <vw/Cartography/CameraBBox.h>
#include <vw/InterestPoint/Matcher.h>
#include <vw/FileIO/KML.h>
#include <asp/Camera/CameraResectioning.h>

#include <string>

using namespace vw;
using namespace vw::camera;
using namespace vw::ba;

void asp::BAParams::record_points_to_kml(const std::string &kml_path,
                                         const vw::cartography::Datum& datum,
                                         size_t skip, const std::string name,
                                         const std::string icon) {
  if (datum.name() == asp::UNSPECIFIED_DATUM) {
    vw::vw_out(vw::WarningMessage) << "No datum specified, can't write file: "
                                   << kml_path << std::endl;
    return;
  }
  
  // Open the file
  vw::vw_out() << "Writing: " << kml_path << std::endl;
  vw::KMLFile kml(kml_path, name);
  
    // Set up a simple point icon with no labels
  const bool hide_labels = true;
  kml.append_style( "point", "", 1.0, icon, hide_labels);
  kml.append_style( "point_highlight", "", 1.1, icon, hide_labels);
  kml.append_stylemap( "point_placemark", "point",
                       "point_highlight");
  
  // Loop through the points
  const bool extrude = true;
  for (size_t i=0; i<num_points(); i+=skip) {
    
    if (get_point_outlier(i))
      continue; // skip outliers
    
    // Convert the point to GDC coords
    vw::Vector3 xyz         = get_point(i);
    vw::Vector3 lon_lat_alt = datum.cartesian_to_geodetic(xyz);

    // Add this to the output file
    kml.append_placemark( lon_lat_alt.x(), lon_lat_alt.y(),
                            "", "", "point_placemark",
                          lon_lat_alt[2], extrude );
  }
    kml.close_kml();
}

void pack_pinhole_to_arrays(vw::camera::PinholeModel const& camera,
                            int camera_index,
                            asp::BAParams & param_storage) {

  double* pos_pose_ptr   = param_storage.get_camera_ptr              (camera_index);
  double* center_ptr     = param_storage.get_intrinsic_center_ptr    (camera_index);
  double* focus_ptr      = param_storage.get_intrinsic_focus_ptr     (camera_index);
  double* distortion_ptr = param_storage.get_intrinsic_distortion_ptr(camera_index);

  // Handle position and pose
  CameraAdjustment pos_pose_info;
  pos_pose_info.copy_from_pinhole(camera);
  pos_pose_info.pack_to_array(pos_pose_ptr);

  // We are solving for multipliers to the intrinsic values, so they all start at 1.0.

  // Center point and focal length
  center_ptr[0] = 1.0; //camera.point_offset()[0];
  center_ptr[1] = 1.0; //camera.point_offset()[1];
  focus_ptr [0] = 1.0; //camera.focal_length()[0];

  // Pack the lens distortion parameters.
  vw::Vector<double> lens = camera.lens_distortion()->distortion_parameters();
  for (size_t i=0; i<lens.size(); ++i)
    distortion_ptr[i] = 1.0;
}

void pack_optical_bar_to_arrays(vw::camera::OpticalBarModel const& camera,
                                int camera_index,
                                asp::BAParams & param_storage) {

  double* pos_pose_ptr   = param_storage.get_camera_ptr              (camera_index);
  double* center_ptr     = param_storage.get_intrinsic_center_ptr    (camera_index);
  double* focus_ptr      = param_storage.get_intrinsic_focus_ptr     (camera_index);
  double* intrinsics_ptr = param_storage.get_intrinsic_distortion_ptr(camera_index);

  // Handle position and pose
  CameraAdjustment pos_pose_info;
  pos_pose_info.copy_from_optical_bar(camera);
  pos_pose_info.pack_to_array(pos_pose_ptr);

  // We are solving for multipliers to the intrinsic values, so they all start at 1.0.

  // Center point and focal length
  center_ptr[0] = 1.0; //camera.point_offset()[0];
  center_ptr[1] = 1.0; //camera.point_offset()[1];
  focus_ptr [0] = 1.0; //camera.focal_length()[0];

  // Pack the speed, MCF, and scan time into the distortion pointer.
  intrinsics_ptr[0] = 1.0;
  intrinsics_ptr[1] = 1.0;
  intrinsics_ptr[2] = 1.0;
}

/// Given a transform with origin at the planet center, like output
/// by pc_align, read the adjustments from cameras_vec, apply this
/// transform on top of them, and write the adjustments back to the vector.
/// - Works for pinhole and non-pinhole case.
void apply_transform_to_cameras(vw::Matrix4x4 const& M, asp::BAParams &param_storage,
                                std::vector<vw::CamPtr>
                                const& cam_ptrs) {

  for (unsigned i = 0; i < param_storage.num_cameras(); i++) {

    // Load the current position/pose of this camera.
    double* cam_ptr = param_storage.get_camera_ptr(i);
    CameraAdjustment cam_adjust(cam_ptr);

    // Create the adjusted camera model
    vw::camera::AdjustedCameraModel cam(cam_ptrs[i], cam_adjust.position(), cam_adjust.pose());
    // Apply the transform
    cam.apply_transform(M);

    // Copy back the adjustments to the camera array.
    cam_adjust.copy_from_adjusted_camera(cam);
    cam_adjust.pack_to_array(cam_ptr);
  }
} // end function apply_transform_to_cameras

// This function takes advantage of the fact that when it is called the cam_ptrs have the same
//  information as is in param_storage!
void apply_transform_to_cameras_pinhole(vw::Matrix4x4 const& M,
                                        asp::BAParams & param_storage,
                                        std::vector<vw::CamPtr>
                                        const& cam_ptrs){

  for (unsigned i = 0; i < param_storage.num_cameras(); i++) {
    // Apply the transform
    boost::shared_ptr<camera::PinholeModel> pin_ptr = 
      boost::dynamic_pointer_cast<vw::camera::PinholeModel>(cam_ptrs[i]);
    pin_ptr->apply_transform(M);

    // Write out to param_storage
    pack_pinhole_to_arrays(*pin_ptr, i, param_storage);    
  }

} // end function apply_transform_to_cameras_pinhole

/// Apply a scale-rotate-translate transform to pinhole cameras and control points
void apply_rigid_transform(vw::Matrix3x3 const & rotation,
                           vw::Vector3   const & translation,
                           double                scale,
                           std::vector<vw::CamPtr> &camera_models,
                           boost::shared_ptr<ControlNetwork> const& cnet) {

  // Apply the transform to the cameras
  for (size_t icam = 0; icam < camera_models.size(); icam++){
    vw::camera::PinholeModel * pincam
      = dynamic_cast<vw::camera::PinholeModel*>(camera_models[icam].get());
    VW_ASSERT(pincam != NULL, vw::ArgumentErr() << "A pinhole camera expected.\n");

    pincam->apply_transform(rotation, translation, scale);
  } // End loop through cameras

  // Apply the transform to all of the world points in the ControlNetwork
  ControlNetwork::iterator iter;
  for (iter=cnet->begin(); iter!=cnet->end(); ++iter) {
    if (iter->type() == ControlPoint::GroundControlPoint)
      continue; // Don't convert the ground control points!

    Vector3 position     = iter->position();
    Vector3 new_position = scale*rotation*position + translation;
    iter->set_position(new_position);
  }
} // End function ApplyRigidTransform


/// Generate a warning if the GCP's are really far from the IP points
/// - This is intended to help catch the common lat/lon swap in GCP files.
void check_gcp_dists(std::vector<vw::CamPtr> const& camera_models,
                     boost::shared_ptr<ControlNetwork> const& cnet_ptr,
                     double forced_triangulation_distance) {

  // Count the points and triangulate
  const ControlNetwork & cnet = *cnet_ptr.get(); // Helper alias
  const int num_cnet_points = static_cast<int>(cnet.size());
  double gcp_count = 0, ip_count = 0;
  Vector3 mean_gcp(0, 0, 0);
  Vector3 mean_ip (0, 0, 0);
  for (int ipt = 0; ipt < num_cnet_points; ipt++) {

    if (cnet[ipt].position() == Vector3() || cnet[ipt].size() <= 1)
      continue;
    
    if (cnet[ipt].type() == ControlPoint::GroundControlPoint) {
      gcp_count += 1.0;
      mean_gcp += cnet[ipt].position();
    } else {
      // Use triangulation to estimate the position of this control point using
      // the current set of camera models.
      ControlPoint cp_new = cnet[ipt];
      double minimum_angle = 0;
      double ans = vw::ba::triangulate_control_point(cp_new, camera_models, minimum_angle,
						     forced_triangulation_distance);
      if (ans < 0 || cp_new.position() == Vector3())
        continue; // Skip points that fail to triangulate

      ip_count += 1.0;
      mean_ip += cp_new.position();
    }
  } // End loop through control network points

  if (ip_count == 0 || gcp_count == 0)
    return; // Can't do this check if we don't have both point types.

  // Average the points
  mean_gcp = mean_gcp / gcp_count;
  mean_ip = mean_ip / ip_count;

  double dist = norm_2(mean_ip - mean_gcp);
  if (dist > 100000)
    vw_out() << "WARNING: GCPs are over 100 km from the other points. Are your lat/lon GCP coordinates swapped?\n";
}

//============================================================================

// Initialize the position and orientation of each pinhole camera model using
// a least squares error transform to match the provided camera positions.
// This function overwrites the camera parameters in-place
bool asp::init_pinhole_model_with_camera_positions
(boost::shared_ptr<ControlNetwork> const& cnet, 
 std::vector<vw::CamPtr> & camera_models,
 std::vector<std::string> const& image_files,
 std::vector<Vector3> const & estimated_camera_gcc) {

  vw_out() << "Initializing camera positions from input file." << std::endl;

  // Count the number of matches and check for problems
  const int num_cameras = image_files.size();
  if (int(estimated_camera_gcc.size()) != num_cameras)
    vw_throw( ArgumentErr() << "No camera matches provided to init function!\n" );

  vw_out() << "Num cameras: " << num_cameras << std::endl;

  int num_matches_found = 0;
  for (int i=0; i<num_cameras; ++i)
    if (estimated_camera_gcc[i] != Vector3(0,0,0))
      ++num_matches_found;

  vw_out() << "Number of matches found: " << num_matches_found << std::endl;

  const int MIN_NUM_MATCHES = 3;
  if (num_matches_found < MIN_NUM_MATCHES)
    vw_throw( ArgumentErr() << "At least " << MIN_NUM_MATCHES 
              << " camera position matches are required to initialize sensor models!\n" );

  // Populate matrices containing the current and known camera positions.
  vw::Matrix<double> points_in(3, num_matches_found), points_out(3, num_matches_found);
  typedef vw::math::MatrixCol<vw::Matrix<double> > ColView;
  int index = 0;
  for (int i=0; i<num_cameras; ++i) {
    // Skip cameras with no matching record
    if (estimated_camera_gcc[i] == Vector3(0,0,0))
      continue;

    // Get the two GCC positions
    Vector3 gcc_in  = camera_models[i]->camera_center(Vector2(0,0));
    Vector3 gcc_out = estimated_camera_gcc[i];

    // Store in matrices
    ColView colIn (points_in,  index); 
    ColView colOut(points_out, index);
    colIn  = gcc_in;
    colOut = gcc_out;
    ++index;

  } // End matrix populating loop

  // Call function to compute a 3D affine transform between the two point sets
  vw::Matrix3x3 rotation;
  vw::Vector3   translation;
  double        scale;
  vw::math::find_3D_transform(points_in, points_out, rotation, translation, scale);

  // Update the camera and point information with the new transform
  apply_rigid_transform(rotation, translation, scale, camera_models, cnet);
  return true;
}

// Given at least two images, each having at least 3 GCP that are not seen in other
// images, find and apply a transform to the camera system based on them.
void asp::transform_cameras_with_indiv_image_gcp
(boost::shared_ptr<ControlNetwork> const& cnet_ptr,
 std::vector<vw::CamPtr> & camera_models) {
  
  vw_out() << "Applying transform to cameras given several GCP not shared among the images.\n";

  int num_cams = camera_models.size();

  // Create pinhole cameras
  std::vector<PinholeModel> pinhole_cams;
  for (int icam = 0; icam < num_cams; icam++){
    vw::camera::PinholeModel * pincam
      = dynamic_cast<vw::camera::PinholeModel*>(camera_models[icam].get());
    VW_ASSERT(pincam != NULL,
	      vw::ArgumentErr() << "A pinhole camera expected.\n");
    pinhole_cams.push_back(*pincam);
  }
  
  // Extract from the control network each pixel for each camera together
  // with its xyz.
  std::vector<std::vector<Vector3>> xyz;
  std::vector<std::vector<Vector2>> pix;
  xyz.resize(num_cams);
  pix.resize(num_cams);
  const ControlNetwork & cnet = *cnet_ptr.get(); // Helper alias

  int ipt = - 1;
  for (auto iter = cnet.begin(); iter != cnet.end(); iter++) {
    ipt++;
    
    // Keep only gcp
    if (cnet[ipt].type() != ControlPoint::GroundControlPoint) {
      continue;
    }
        
    for (auto measure = (*iter).begin(); measure != (*iter).end(); measure++) {
      int cam_it = measure->image_id();
      if (cam_it < 0 || cam_it >= num_cams) 
	vw_throw(ArgumentErr() << "Error: cnet index out of range.\n");

      Vector2 pixel( measure->position()[0],  measure->position()[1]);
      pix[cam_it].push_back(pixel);
      xyz[cam_it].push_back(cnet[ipt].position());
    }
  }  

  Matrix3x3 rotation;
  Vector3   translation;
  double    scale;
  asp::align_cameras_to_ground(xyz, pix, pinhole_cams, rotation, translation, scale);

  // Update the camera and point information with the new transform
  vw_out() << "Applying transform based on GCP:\n";
  vw_out() << "Rotation:    " << rotation    << "\n";
  vw_out() << "Translation: " << translation << "\n";
  vw_out() << "Scale:       " << scale       << "\n";
  apply_rigid_transform(rotation, translation, scale, camera_models, cnet_ptr);
}

/// Initialize the position and orientation of each pinhole camera model using
/// a least squares error transform to match the provided control points file.
/// This function overwrites the camera parameters in-place. It works
/// if at least three GCP are seen in no less than two images.
void asp::transform_cameras_with_shared_gcp(boost::shared_ptr<ControlNetwork> const& cnet_ptr,
                                            std::vector<vw::CamPtr> & camera_models) {
  
  vw_out() << "Applying transform to cameras given several GCP shared among the images.\n";

  const ControlNetwork & cnet = *cnet_ptr.get(); // Helper alias
  
  // Verify that all cameras are pinhole
  for (size_t icam = 0; icam < camera_models.size(); icam++){
    vw::camera::PinholeModel * pincam
      = dynamic_cast<vw::camera::PinholeModel*>(camera_models[icam].get());
    VW_ASSERT(pincam != NULL,
	      vw::ArgumentErr() << "A pinhole camera expected.\n");
  }
  
  // Put the good ground control points in a vector.
  int num_cnet_points = static_cast<int>(cnet.size());
  std::vector<vw::Vector3> in_xyz, out_xyz; 
  int num_gcp      = 0;
  int num_good_gcp = 0;
  for (int ipt = 0; ipt < num_cnet_points; ipt++) {
    
    if (cnet[ipt].type() != ControlPoint::GroundControlPoint)
      continue;

    num_gcp++;
    
    // Use triangulation to estimate the position of this control point using
    //   the current set of camera models.
    ControlPoint untrans_cp = cnet[ipt];
    double minimum_angle = 1.0e-3; // Likely this is too small for a good GCP, but better than 0.
    double forced_triangulation_distance = -1.0;
    double err = vw::ba::triangulate_control_point(untrans_cp, camera_models,
						   minimum_angle, forced_triangulation_distance);
    
    if (untrans_cp.position() != Vector3() &&
        cnet[ipt].position()  != Vector3() &&
	err >= 0) {
      // Store the computed and correct position of this point
      in_xyz.push_back(untrans_cp.position());
      out_xyz.push_back(cnet[ipt].position());
      num_good_gcp++; // Only count points that triangulate
    } else {
      vw_out() << "Discarding GCP that could not be triangulated: " << cnet[ipt] << ".\n";
    }
  } // End good GCP counting

  // Sanity check
  const int MIN_GCP_COUNT = 3;
  if (num_good_gcp < MIN_GCP_COUNT) {
    vw_out() << "Num GCP       = " << num_gcp      << std::endl;
    vw_out() << "Num valid GCP = " << num_good_gcp << std::endl;
    vw_throw( ArgumentErr()
	      << "Not enough valid GCPs to apply a transform to the cameras. "
	      << "You may need to use --transform-cameras-using-gcp.\n" );
  }

  // Copy these points to a matrix as required by the API about to be used. 
  vw::Matrix<double> points_in(3, num_good_gcp), points_out(3, num_good_gcp);
  typedef vw::math::MatrixCol<vw::Matrix<double>> ColView;
  for (size_t ipt = 0; ipt < in_xyz.size(); ipt++) {
    ColView colIn (points_in,  ipt); 
    ColView colOut(points_out, ipt);
    colIn  = in_xyz[ipt];
    colOut = out_xyz[ipt];
  } // End loop through control network points
  
  // Call function to compute a 3D affine transform between the two point sets
  vw::Matrix3x3 rotation;
  vw::Vector3   translation;
  double        scale;
  vw::math::find_3D_transform(points_in, points_out, rotation, translation, scale);
  
  // Update the camera and point information with the new transform
  vw_out() << "Applying transform based on GCP:\n";
  vw_out() << "Rotation:    " << rotation    << "\n";
  vw_out() << "Translation: " << translation << "\n";
  vw_out() << "Scale:       " << scale       << "\n";
  vw_out() << "This transform can be disabled with --disable-pinhole-gcp-init.\n";
  apply_rigid_transform(rotation, translation, scale, camera_models, cnet_ptr);

  return;
} // End function transform_cameras_with_shared_gcp

/// Initialize the position and orientation of a pinhole camera model using
/// GCP. It invokes OpenCV's PnP functionality.
void asp::init_camera_using_gcp(boost::shared_ptr<vw::ba::ControlNetwork> const& cnet_ptr,
                                std::vector<vw::CamPtr> & camera_models) {
  
  // Sanity check
  if (camera_models.size() != 1) 
    vw::vw_throw(vw::ArgumentErr() << "Cannot initialize more than a camera at a time using GCP. "
                 << "Consider using --transform-cameras-with-shared-gcp or "
                 << "--transform-cameras-using-gcp.\n");
  
  vw_out() << "Initializing a Pinhole camera using GCP.\n";

  int icam = 0;
  vw::camera::PinholeModel * pincam
    = dynamic_cast<vw::camera::PinholeModel*>(camera_models[icam].get());
  VW_ASSERT(pincam != NULL, vw::ArgumentErr() << "A pinhole camera expected.\n");
  
  std::vector<vw::Vector2> pixel_observations;
  std::vector<vw::Vector3> ground_points;
  const ControlNetwork & cnet = *cnet_ptr.get(); // Helper alias
  for (int ipt = 0; ipt < cnet.size(); ipt++){

    // Loop through all the ground control points only
    if (cnet[ipt].type() != ControlPoint::GroundControlPoint)
      continue;
    ground_points.push_back(cnet[ipt].position());
    
    int num_meas = 0;
    for (ControlPoint::const_iterator measure = cnet[ipt].begin();
         measure != cnet[ipt].end(); measure++) {
      
      int cam_it = measure->image_id();
      if (cam_it != 0) 
        vw_throw(ArgumentErr() << "Error: Expecting GCP for a single camera.\n");
      
      Vector2 pixel(measure->position()[0], measure->position()[1]);
      num_meas++;
      if (num_meas > 1)
        vw::vw_throw(vw::ArgumentErr() << "Expecting a single camera pixel per gcp.\n");
      
      pixel_observations.push_back(pixel);
    }
  }

  // Update the camera pose with given observations and intrinsics
  asp::findCameraPose(ground_points, pixel_observations, *pincam);

  return;
  
} // End function init_camera_using_gcp

// Given original cams in sfm_cams and individually scaled cameras in
// aux_cams, get the median scale change from the first set to the second one.
// It is important to do the median, since scaling the cameras individually
// is a bit of a shaky business.
double asp::find_median_scale_change(std::vector<PinholeModel> const & sfm_cams,
                                     std::vector<PinholeModel> const & aux_cams,
                                     std::vector< std::vector<Vector3> > const& xyz){
  
  int num_cams = sfm_cams.size();

  std::vector<double> scales;
  
  for (int it1 = 0; it1 < num_cams; it1++) {

    bool is_good = (xyz[it1].size() >= 3);
    if (!is_good)
      continue;
    
    for (int it2 = it1 + 1; it2 < num_cams; it2++) {
      
      bool is_good = (xyz[it2].size() >= 3);
      if (!is_good)
	continue;
    
      double len1 = norm_2(sfm_cams[it1].camera_center()
			   - sfm_cams[it2].camera_center());
      double len2 = norm_2(aux_cams[it1].camera_center()
			   - aux_cams[it2].camera_center());
      
      double scale = len2/len1;
      scales.push_back(scale);
    }
  }

  if (scales.empty())
    vw_throw( LogicErr() << "Could not find two images with at least 3 GCP each.\n");
    
  double median_scale = vw::math::destructive_median(scales);

  return median_scale;
}


// Given some GCP so that at least two images have at at least three GCP each,
// but each GCP is allowed to show in one image only, use the GCP
// to transform cameras to ground coordinates.
void asp::align_cameras_to_ground(std::vector< std::vector<Vector3> > const& xyz,
                                  std::vector< std::vector<Vector2> > const& pix,
                                  std::vector<PinholeModel> & sfm_cams,
                                  Matrix3x3 & rotation, 
                                  Vector3 & translation,
                                  double & scale){
  
  std::string camera_type = "pinhole";
  bool refine_camera = true;
  bool verbose = false; 

  // Cameras individually aligned to ground using GCP. They may not be
  // self-consistent, and are only used to give an idea of the
  // transform to apply to the unaligned cameras.
  std::vector<PinholeModel> aux_cams;

  int num_cams = sfm_cams.size();
  for (int it = 0; it < num_cams; it++) {
    // Export to the format used by the API
    std::vector<double> pixel_values;
    for (size_t c = 0; c < pix[it].size(); c++) {
      pixel_values.push_back(pix[it][c][0]);
      pixel_values.push_back(pix[it][c][1]);
    }

    vw::CamPtr out_cam(new PinholeModel(sfm_cams[it]));

    bool is_good = (xyz[it].size() >= 3);
    if (is_good) 
      fit_camera_to_xyz(camera_type, refine_camera,  
			xyz[it], pixel_values, verbose, out_cam);
    
    aux_cams.push_back(*((PinholeModel*)out_cam.get()));
  }

  double world_scale = asp::find_median_scale_change(sfm_cams, aux_cams, xyz);
  vw_out() << "Initial guess scale to apply when converting to world coordinates using GCP: "
	   << world_scale << ".\n";

  // So far we aligned both cameras individually to GCP and we got an
  // idea of scale.  Yet we would like to align them without changing
  // the relationship between them, so using a single transform for
  // all not an individual transform for each.  This way we will
  // transform the SfM-computed cameras to the new coordinate system.

  // Start by estimating a such a transform.
  int num_pts = 0;
  for (int it = 0; it < num_cams; it++) {
    bool is_good = (xyz[it].size() >= 3);
    if (is_good) 
      num_pts += pix[it].size();
  }
  
  vw::Matrix<double> in_pts, out_pts;
  in_pts.set_size(3, num_pts);
  out_pts.set_size(3, num_pts);
  
  int col = 0;
  for (int it = 0; it < num_cams; it++) {
    
    bool is_good = (xyz[it].size() >= 3);
    if (is_good) {
      // For each camera, find xyz values in the input cameras
      // that map to GCP. Use the scale for that.
      for (int c = 0; c < xyz[it].size(); c++) {
	
	// Distance from camera center to xyz for the individually aligned cameras
	double len = norm_2(aux_cams[it].camera_center() - xyz[it][c]);
	len = len / world_scale;
	Vector3 trans_xyz = sfm_cams[it].camera_center()
	  + len * sfm_cams[it].pixel_to_vector(pix[it][c]);
	for (int row = 0; row < in_pts.rows(); row++) {
	  in_pts(row, col)  = trans_xyz[row];
	  out_pts(row, col) = xyz[it][c][row];
	}
	
	col++;
      }
    }
  }
  
  if (col != num_pts) 
    vw_throw( LogicErr() << "Book-keeping failure in aligning cameras to ground.\n");

  // The initial transform to world coordinates
  Vector<double> C;
  vw::math::find_3D_transform(in_pts, out_pts, rotation, translation, scale);

  // Copy into C
  transform_to_vector(C, rotation, translation, scale);

  // Form the pixel vector
  int pixel_vec_len = 0;
  for (size_t it = 0; it < pix.size(); it++) {
    bool is_good = (xyz[it].size() >= 3);
    if (is_good)
      pixel_vec_len += pix[it].size() * 2;
  }
  Vector<double> pixel_vec;
  pixel_vec.set_size(pixel_vec_len);
  int count = 0;
  for (size_t it = 0; it < pix.size(); it++) {
    bool is_good = (xyz[it].size() >= 3);
    if (is_good) {
      for (size_t c = 0; c < pix[it].size(); c++) {
	Vector2 pixel = pix[it][c];
	pixel_vec[2*count  ] = pixel[0];
	pixel_vec[2*count+1] = pixel[1];
	count++;
      }
    }
  }
  if (2*count != pixel_vec_len)
    vw_throw( LogicErr() << "Book-keeping failure in cam_gen.\n");
  
  // Optimize the transform
  double abs_tolerance  = 1e-24;
  double rel_tolerance  = 1e-24;
  int    max_iterations = 2000;
  int status = 0;
  CameraSolveRotTransScale<PinholeModel> lma_model(xyz, pixel_vec, sfm_cams);
  Vector<double> final_params
    = vw::math::levenberg_marquardt(lma_model, C, pixel_vec,
				    status, abs_tolerance, rel_tolerance,
				    max_iterations);

  Vector<double>  final_residual = lma_model(final_params, verbose);
  
  // Bring the cameras to world coordinates
  for (int it = 0; it < num_cams; it++) 
    apply_rot_trans_scale(sfm_cams[it], final_params);

  // Unpack the final vector into a rotation + translation + scale
  vector_to_transform(final_params, rotation, translation, scale);

}

/// Take an interest point from a map projected image and convert it
/// to the corresponding IP in the original non-map-projected image.
/// - Return false if the pixel could not be converted.
bool asp::projected_ip_to_raw_ip(vw::ip::InterestPoint &P,
                                 vw::ImageViewRef<vw::PixelMask<double>> const& interp_dem,
                                 vw::CamPtr camera_model,
                                 vw::cartography::GeoReference const& georef,
                                 vw::cartography::GeoReference const& dem_georef) {
  // Get IP coordinate in the DEM
  Vector2 pix(P.x, P.y);
  Vector2 ll      = georef.pixel_to_lonlat(pix);
  Vector2 dem_pix = dem_georef.lonlat_to_pixel(ll);
  if (!interp_dem.pixel_in_bounds(dem_pix))
    return false;
  // Load the elevation from the DEM
  PixelMask<double> dem_val = interp_dem(dem_pix[0], dem_pix[1]);
  if (!is_valid(dem_val))
    return false;
  Vector3 llh(ll[0], ll[1], dem_val.child());
  Vector3 xyz = dem_georef.datum().geodetic_to_cartesian(llh);

  // Project into the camera
  Vector2 cam_pix;
  try {
   cam_pix = camera_model->point_to_pixel(xyz);
  } catch(...) {
    return false; // Don't update the point.
  }
  P.x  = cam_pix.x();
  P.y  = cam_pix.y();
  P.ix = P.x;
  P.iy = P.y;
  return true;
}

// This function takes advantage of the fact that when it is called the cam_ptrs have the same
//  information as is in param_storage!
void apply_transform_to_cameras_optical_bar(vw::Matrix4x4 const& M,
                                            asp::BAParams & param_storage,
                                            std::vector<vw::CamPtr> const& cam_ptrs){

  // Convert the transform format
  vw::Matrix3x3 R = submatrix(M, 0, 0, 3, 3);
  vw::Vector3   T;
  for (int r = 0; r < 3; r++) 
    T[r] = M(r, 3);
  
  double scale = pow(det(R), 1.0/3.0);
  for (size_t r = 0; r < R.rows(); r++)
    for (size_t c = 0; c < R.cols(); c++)
      R(r, c) /= scale;

  for (unsigned i = 0; i < param_storage.num_cameras(); i++) {

    // Apply the transform
    boost::shared_ptr<vw::camera::OpticalBarModel> bar_ptr = 
      boost::dynamic_pointer_cast<vw::camera::OpticalBarModel>(cam_ptrs[i]);
    bar_ptr->apply_transform(R, T, scale);

    // Write out to param_storage
    pack_optical_bar_to_arrays(*bar_ptr, i, param_storage);    
  }

} // end function apply_transform_to_cameras_pinhole

// Given an input pinhole camera and param changes, apply those, returning
// the new camera. Note that all intrinsic parameters are stored as multipliers
// in asp::BAParams.
vw::camera::PinholeModel transformedPinholeCamera(int camera_index,
                                                  asp::BAParams const& param_storage,
                                                  vw::camera::PinholeModel const& in_cam) {

  // Start by making a copy of the camera. Note that this does not make a copy of the
  // distortion params, as that's a pointer. So will have to make a copy of it further down.
  vw::camera::PinholeModel out_cam = in_cam;

  double const* pos_pose_ptr   = param_storage.get_camera_ptr(camera_index);
  double const* center_ptr     = param_storage.get_intrinsic_center_ptr    (camera_index);
  double const* focus_ptr      = param_storage.get_intrinsic_focus_ptr     (camera_index);
  double const* distortion_ptr = param_storage.get_intrinsic_distortion_ptr(camera_index);

  // Update position and pose
  CameraAdjustment pos_pose_info(pos_pose_ptr);
  out_cam.set_camera_center(pos_pose_info.position());
  out_cam.set_camera_pose  (pos_pose_info.pose    ());

  // Update the lens distortion parameters. Note how we make a new copy of the distortion object.
  boost::shared_ptr<LensDistortion> distortion = out_cam.lens_distortion()->copy();
  vw::Vector<double> lens = distortion->distortion_parameters();
  for (size_t i=0; i<lens.size(); ++i)
    lens[i] *= distortion_ptr[i];
  distortion->set_distortion_parameters(lens);
  out_cam.set_lens_distortion(distortion.get());

  // Update the center and focus
  Vector2 old_center = out_cam.point_offset();
  Vector2 old_focus  = out_cam.focal_length();
  out_cam.set_point_offset(Vector2(center_ptr[0]*old_center[0],
                                  center_ptr[1]*old_center[1]), false);
  double new_focus = old_focus[0]*focus_ptr[0];
  out_cam.set_focal_length(Vector2(new_focus,new_focus), true); // Recompute internals.
  
  return out_cam;
}

// Given an input optical bar camera and param changes, apply those, returning
// the new camera.
vw::camera::OpticalBarModel transformedOpticalBarCamera(int camera_index,
                                                        asp::BAParams const& param_storage,
                                                        vw::camera::OpticalBarModel const& in_cam) {
  
  // Start by making a copy of the camera.
  vw::camera::OpticalBarModel out_cam = in_cam;

  double const* pos_pose_ptr  = param_storage.get_camera_ptr(camera_index);
  double const* center_ptr    = param_storage.get_intrinsic_center_ptr    (camera_index);
  double const* focus_ptr     = param_storage.get_intrinsic_focus_ptr     (camera_index);
  double const* intrinsic_ptr = param_storage.get_intrinsic_distortion_ptr(camera_index);

  // Update position and pose
  CameraAdjustment pos_pose_info(pos_pose_ptr);
  out_cam.set_camera_center(pos_pose_info.position());
  out_cam.set_camera_pose  (pos_pose_info.pose    ());

  // All intrinsic parameters are stored as multipliers!

  // Update the other intrinsic parameters.
  out_cam.set_speed              (out_cam.get_speed()*intrinsic_ptr[0]);
  out_cam.set_motion_compensation(out_cam.get_motion_compensation()*intrinsic_ptr[1]);
  out_cam.set_scan_time          (out_cam.get_scan_time()*intrinsic_ptr[2]);

  // Update the center and focus
  Vector2 old_center = out_cam.get_optical_center();
  float   old_focus  = out_cam.get_focal_length();
  out_cam.set_optical_center(Vector2(center_ptr[0]*old_center[0],
                                    center_ptr[1]*old_center[1]));
  double new_focus = old_focus*focus_ptr[0];
  out_cam.set_focal_length(new_focus);

  return out_cam;
}


// Save convergence angle percentiles for each image pair having matches
void asp::saveConvergenceAngles(std::string const& conv_angles_file,
                                std::vector<asp::MatchPairStats> const& convAngles,
                                std::vector<std::string> const& imageFiles) {

  vw_out() << "Writing: " << conv_angles_file << "\n";
  std::ofstream ofs (conv_angles_file.c_str());
  ofs.precision(8);
  ofs << "# Convergence angle percentiles (in degrees) for each image pair having matches\n";
  ofs << "# left_image right_image 25% 50% 75% num_angles_per_pair\n";
  for (size_t conv_it = 0; conv_it < convAngles.size(); conv_it++) {
    auto const & c = convAngles[conv_it]; // alias
    ofs << imageFiles[c.left_cam_index] << ' ' << imageFiles[c.right_cam_index] << ' '
        << c.val25 << ' ' << c.val50 << ' ' << c.val75  << ' ' << c.num_vals << "\n";
  }
  ofs.close();

  return;
}

// Mapproject interest points onto a DEM and find the norm of their
// disagreement in meters. It is assumed that dem_georef
// was created by bilinear interpolation. The cameras must be with
// the latest adjustments applied to them.
void asp::calcPairMapprojOffsets(int left_cam_index, int right_cam_index,
                                 std::vector<vw::CamPtr>            const& optimized_cams,
                                 std::vector<vw::ip::InterestPoint> const& left_ip,
                                 std::vector<vw::ip::InterestPoint> const& right_ip,
                                 vw::cartography::GeoReference      const& dem_georef,
                                 vw::ImageViewRef<vw::PixelMask<double>>  & interp_dem,
                                 // Will append below
                                 std::vector<vw::Vector<float, 4>>       & mapprojPoints,
                                 std::vector<float>                      & mapprojOffsets) {
  
  // Wipe mapprojOffsets
  mapprojOffsets.clear();
  // Will append to mapprojPoints, so don't wipe it
  
  for (size_t ip_it = 0; ip_it < left_ip.size(); ip_it++) {
    
    bool treat_nodata_as_zero = false;
    bool has_intersection = false;
    double height_error_tol = 0.001; // 1 mm should be enough
    double max_abs_tol      = 1e-14; // abs cost fun change b/w iterations
    double max_rel_tol      = 1e-14;
    int num_max_iter        = 50;   // Using many iterations can be very slow
    Vector3 xyz_guess;
    
    Vector2 left_pix(left_ip[ip_it].x, left_ip[ip_it].y);
    Vector3 left_dem_xyz = vw::cartography::camera_pixel_to_dem_xyz
      (optimized_cams[left_cam_index]->camera_center(left_pix),
       optimized_cams[left_cam_index]->pixel_to_vector(left_pix),
       interp_dem, dem_georef, treat_nodata_as_zero, has_intersection,
       height_error_tol, max_abs_tol, max_rel_tol, num_max_iter, xyz_guess);
    if (!has_intersection) 
      continue;
    
    // Do the same for right. Use left pixel as initial guess
    xyz_guess = left_dem_xyz;
    Vector2 right_pix(right_ip[ip_it].x, right_ip[ip_it].y);
    Vector3 right_dem_xyz = vw::cartography::camera_pixel_to_dem_xyz
      (optimized_cams[right_cam_index]->camera_center(right_pix),
       optimized_cams[right_cam_index]->pixel_to_vector(right_pix),
       interp_dem, dem_georef, treat_nodata_as_zero, has_intersection,
       height_error_tol, max_abs_tol, max_rel_tol, num_max_iter, xyz_guess);
    if (!has_intersection) 
      continue;

    Vector3 mid_pt = (left_dem_xyz + right_dem_xyz)/2.0;
    double dist = norm_2(left_dem_xyz - right_dem_xyz);

    // Keep in the same structure both the midpoint between these two
    // mapprojected ip, as lon-lat,height, and their distance, as
    // later the bookkeeping of mapprojOffsets will be different.
    // Float precision is enough, and will save on memory.
    Vector<float, 4> point;
    subvector(point, 0, 3) = dem_georef.datum().cartesian_to_geodetic(mid_pt);
    point[3] = dist;
    
    mapprojPoints.push_back(point);
    mapprojOffsets.push_back(dist);
  }
}

// Save mapprojected matches offsets for each image pair having matches
void asp::saveMapprojOffsets(std::string                       const& mapproj_offsets_stats_file,
                             std::string                       const& mapproj_offsets_file,
                             vw::cartography::GeoReference     const& mapproj_dem_georef,
                             std::vector<vw::Vector<float, 4>> const& mapprojPoints,
                             std::vector<asp::MatchPairStats>  const& mapprojOffsets,
                             std::vector<std::vector<float>>        & mapprojOffsetsPerCam,
                             std::vector<std::string>          const& imageFiles) {
  
  vw_out() << "Writing: " << mapproj_offsets_stats_file << "\n";
  std::ofstream ofs (mapproj_offsets_stats_file.c_str());
  ofs.precision(8); // 8 digits of precision for errors is enough

  ofs << "# Percentiles of distances between mapprojected matching pixels in an "
      << "image and the others.\n";
  ofs << "# image_name 25% 50% 75% 85% 95% count\n";
  for (size_t image_it = 0; image_it < imageFiles.size(); image_it++) {
    auto & vals = mapprojOffsetsPerCam[image_it]; // alias
    int len = vals.size();
    float val25 = -1.0, val50 = -1.0, val75 = -1.0, val85 = -1.0, val95 = -1.0, count = 0;
    if (!vals.empty()) {
      std::sort(vals.begin(), vals.end());
      val25 = vals[0.25 * len];
      val50 = vals[0.50 * len];
      val75 = vals[0.75 * len];
      val85 = vals[0.85 * len];
      val95 = vals[0.95 * len];
      count = len;
    }

    ofs << imageFiles[image_it] << ' '
        << val25 << ' ' << val50 << ' ' << val75 << ' '
        << val85 << ' ' << val95 << ' ' << count << "\n";
  }

  ofs << "# Percentiles of distances between matching pixels after mapprojecting onto DEM.\n"
      << "# Per image pair and measured in DEM pixel units.\n";
  ofs << "# left_image right_image 25% 50% 75% 85% 95% num_matches_per_pair\n";
  ofs.precision(8); // 8 digits of precision for errors is enough
  for (size_t conv_it = 0; conv_it < mapprojOffsets.size(); conv_it++) {
    auto const & c = mapprojOffsets[conv_it]; // alias
    ofs << imageFiles[c.left_cam_index] << ' ' << imageFiles[c.right_cam_index] << ' '
        << c.val25 << ' ' << c.val50 << ' ' << c.val75 << ' '
        << c.val85 << ' ' << c.val95 << ' ' << c.num_vals << "\n";
  }

  ofs.close();

  vw_out() << "Writing: " << mapproj_offsets_file << "\n";
  ofs = std::ofstream(mapproj_offsets_file.c_str());
  ofs.precision(8); // 8 digits of precision for errors is enough
  ofs << "# lon, lat, height_above_datum, mapproj_ip_dist_meters\n";
  ofs << "# " << mapproj_dem_georef.datum() << std::endl;

  // Write all the points to the file
  for (size_t it = 0; it < mapprojPoints.size(); it++) {
    Vector3 llh = subvector(mapprojPoints[it], 0, 3);
    ofs << llh[0] << ", " << llh[1] <<", " << llh[2] << ", "
         << mapprojPoints[it][3] << std::endl;
  }
  
  ofs.close();
  
  return;
}

// Calculate convergence angles. Remove the outliers flagged earlier,
// if remove_outliers is true. Compute offsets of mapprojected matches,
// if a DEM is given. These are done together as they rely on
// reloading interest point matches, which is expensive so the matches
// are used for both operations.
void asp::matchFilesProcessing(vw::ba::ControlNetwork       const& cnet,
                               asp::BaBaseOptions           const& opt,
                               std::vector<vw::CamPtr>      const& optimized_cams,
                               bool                                remove_outliers,
                               std::set<int>                const& outliers,
                               std::string                  const& mapproj_dem,
                               std::vector<asp::MatchPairStats>  & convAngles,
                               std::vector<vw::Vector<float, 4>> & mapprojPoints,
                               std::vector<asp::MatchPairStats>  & mapprojOffsets,
                               std::vector<std::vector<float>>   & mapprojOffsetsPerCam) {

  vw_out() << "Filtering outliers and creating reports.\n";
  
  mapprojPoints.clear();
  convAngles.clear();
  mapprojOffsets.clear();
  mapprojOffsetsPerCam.clear();

  bool save_mapproj_match_points_offsets = (!mapproj_dem.empty());
  vw::cartography::GeoReference mapproj_dem_georef;
  ImageViewRef<PixelMask<double>> interp_mapproj_dem;
  if (save_mapproj_match_points_offsets)
    asp::create_interp_dem(mapproj_dem, mapproj_dem_georef, interp_mapproj_dem);

  int num_cameras = opt.image_files.size();
  mapprojOffsetsPerCam.resize(num_cameras);

  // Iterate over the control network, and, for each inlier pair of matches,
  // remember what pair it is from. Needed only if there is outlier filtering.
  // TODO(oalexan1): This uses a lot of memory. Need to keep just indices, somehow, not
  // quadruplets of floats.
  typedef std::tuple<float, float, float, float> Quadruplet;
  std::map<std::pair<int, int>, std::set<Quadruplet>> inlier_pairs;
  if (remove_outliers) {
    int ipt = -1;
    for (ControlNetwork::const_iterator iter = cnet.begin(); iter != cnet.end(); iter++) {
      // Control point index
      ipt++; 
      // Skip outliers
      if (outliers.find(ipt) != outliers.end())
        continue;
      // Skip gcp
      if (cnet[ipt].type() == ControlPoint::GroundControlPoint)
        continue;
      
      for (auto m1 = (*iter).begin(); m1 != (*iter).end(); m1++) {
        for (auto m2 = (*iter).begin(); m2 != (*iter).end(); m2++) {
          int left_index = m1->image_id();
          int right_index = m2->image_id();
          if (left_index >= right_index) 
            continue;
          inlier_pairs[std::make_pair(left_index, right_index)].insert
            (Quadruplet(m1->position()[0], m1->position()[1],
                        m2->position()[0], m2->position()[1]));
        }
      }
    }
  }
  
  // Work on individual image pairs
  for (auto match_it = opt.match_files.begin(); match_it != opt.match_files.end(); match_it++) {

    std::pair<int, int> cam_pair   = match_it->first;
    std::string         match_file = match_it->second;
    size_t left_index  = cam_pair.first;
    size_t right_index = cam_pair.second;
    if (left_index >= right_index) 
      vw::vw_throw(vw::ArgumentErr() << "Bookkeeping failure. Left image index "
                   << "must be less than right image index.\n");
    
    // Just skip over match files that don't exist.
    if (!boost::filesystem::exists(match_file)) {
      vw_out() << "Skipping non-existent match file: " << match_file << std::endl;
      continue;
    }

    // Read the original IP, to ensure later we write to disk only
    // the subset of the IP from the control network which
    // are part of these original ones. 
    std::vector<ip::InterestPoint> orig_left_ip, orig_right_ip;
    ip::read_binary_match_file(match_file, orig_left_ip, orig_right_ip);

    // Create a new convergence angle storage struct
    convAngles.push_back(asp::MatchPairStats()); // add an element, will populate it soon
    asp::MatchPairStats & convAngle = convAngles.back(); // alias
    std::vector<double> sorted_angles;
    if (save_mapproj_match_points_offsets)
      mapprojOffsets.push_back(asp::MatchPairStats()); // add an elem

    if (!remove_outliers) {
      asp::convergence_angles(optimized_cams[left_index].get(), optimized_cams[right_index].get(),
                              orig_left_ip, orig_right_ip, sorted_angles);
      convAngle.populate(left_index, right_index, sorted_angles);

      if (save_mapproj_match_points_offsets) {
        std::vector<float> localMapprojOffsets;
        asp::calcPairMapprojOffsets(left_index, right_index,
                                    optimized_cams,
                                    orig_left_ip, orig_right_ip,
                                    mapproj_dem_georef, interp_mapproj_dem,  
                                    mapprojPoints, // will append here
                                    localMapprojOffsets);
        mapprojOffsets.back().populate(left_index, right_index, localMapprojOffsets);
        for (size_t map_it = 0; map_it < localMapprojOffsets.size(); map_it++) {
          mapprojOffsetsPerCam[left_index].push_back(localMapprojOffsets[map_it]);
          mapprojOffsetsPerCam[right_index].push_back(localMapprojOffsets[map_it]);
        }
      }
      
      // Since no outliers are removed, nothing else to do
      continue;
    }

    // Keep only inliers and non-gcp. GCP are used in optimization but are not
    // part of the originally found interest point matches.
    std::vector<vw::ip::InterestPoint> left_ip, right_ip;
    for (size_t ip_iter = 0; ip_iter < orig_left_ip.size(); ip_iter++) {
      Quadruplet q(orig_left_ip[ip_iter].x, orig_left_ip[ip_iter].y,
                   orig_right_ip[ip_iter].x, orig_right_ip[ip_iter].y);
      auto & inlier_pair = inlier_pairs[std::make_pair(left_index, right_index)]; // alias
      if (inlier_pair.find(q) == inlier_pair.end()) 
        continue;

      // We do not copy descriptors, those take storage
      left_ip.push_back(ip::InterestPoint(orig_left_ip[ip_iter].x, orig_left_ip[ip_iter].y,
                                          orig_left_ip[ip_iter].scale));
      right_ip.push_back(ip::InterestPoint(orig_right_ip[ip_iter].x, orig_right_ip[ip_iter].y,
                                           orig_right_ip[ip_iter].scale));
    }
    
    // Filter by disparity
    // TODO(oalexan1): Note that this does not update the outliers set. Likely this
    // processing needs to move where other outlier filtering logic is.
    bool quiet = true; // Otherwise too many messages are printed
    if (opt.remove_outliers_params[0] > 0 && opt.remove_outliers_params[1] > 0.0) {
      // The typical value of 75 for opt.remove_outliers_params[1] may be too low.
      // Adjust it. pct = 75 becomes pct = 90. pct = 100 becomes pct = 100. So,
      // if starting under 100, it gets closer to 100 but stays under it.
      double pct = opt.remove_outliers_params[0];
      pct = 100.0 * (pct + 150.0) / 250.0;
      asp::filter_ip_by_disparity(pct, opt.remove_outliers_params[1],
                                  quiet, left_ip, right_ip);
    }
    
    if (num_cameras == 2){
      // Compute the coverage fraction
      Vector2i right_image_size = file_image_size(opt.image_files[1]);
      int right_ip_width = right_image_size[0]*
        static_cast<double>(100.0 - std::max(opt.ip_edge_buffer_percent, 0))/100.0;
      Vector2i ip_size(right_ip_width, right_image_size[1]);
      double ip_coverage = asp::calc_ip_coverage_fraction(right_ip, ip_size);
      // Careful with the line below, it gets used in process_icebridge_batch.py.
      vw_out() << "IP coverage fraction after cleaning = " << ip_coverage << "\n";
    }

    // Make a clean copy of the file
    std::string clean_match_file = ip::clean_match_filename(match_file);
    if (opt.clean_match_files_prefix != "") {
      // Avoid saving clean-clean.match.
      clean_match_file = match_file;
      // Write the clean match file in the current dir, not where it was read from
      clean_match_file.replace(0, opt.clean_match_files_prefix.size(), opt.out_prefix);
    }
    else if (opt.match_files_prefix != "") {
      // Write the clean match file in the current dir, not where it was read from
      clean_match_file.replace(0, opt.match_files_prefix.size(), opt.out_prefix);
    }
    
    vw_out() << "Saving " << left_ip.size() << " filtered interest points.\n";

    vw_out() << "Writing: " << clean_match_file << std::endl;
    ip::write_binary_match_file(clean_match_file, left_ip, right_ip);

    // Find convergence angles based on clean ip
    asp::convergence_angles(optimized_cams[left_index].get(), optimized_cams[right_index].get(),
                            left_ip, right_ip, sorted_angles);
    convAngle.populate(left_index, right_index, sorted_angles);
    
    if (save_mapproj_match_points_offsets) {
      std::vector<float> localMapprojOffsets;
      asp::calcPairMapprojOffsets(left_index, right_index,
                                  optimized_cams,
                                  left_ip, right_ip,
                                  mapproj_dem_georef, interp_mapproj_dem,  
                                  mapprojPoints, // will append here
                                  localMapprojOffsets);
      mapprojOffsets.back().populate(left_index, right_index, localMapprojOffsets);
      for (size_t map_it = 0; map_it < localMapprojOffsets.size(); map_it++) {
        mapprojOffsetsPerCam[left_index].push_back(localMapprojOffsets[map_it]);
        mapprojOffsetsPerCam[right_index].push_back(localMapprojOffsets[map_it]);
      }
    }
    
  } // End loop through the match files
}
