#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Camera/CameraSolve.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Camera/SPOT_XML.h>
#include <asp/Camera/LinescanSpotModel.h>

namespace asp {

using vw::Vector3;
using vw::Matrix3x3;

// TODO: Port these changes to the base class

vw::Vector2 SPOTCameraModel::point_to_pixel(Vector3 const& point, double starty) const {

  // Use the generic solver to find the pixel 
  // - This method will be slower but works for more complicated geometries
  vw::camera::CameraGenericLMA model( this, point );
  int status;
  vw::Vector2 start = m_image_size / 2.0; // Use the center as the initial guess
  if (starty >= 0) // If the user provided a line number guess..
    start[1] = starty;

  // Solver constants
  const double ABS_TOL = 1e-16;
  const double REL_TOL = 1e-16;
  const int    MAX_ITERATIONS = 1e+5;
  const double MAX_ERROR = 0.01;

  Vector3 objective(0, 0, 0);
  vw::Vector2 solution = vw::math::levenberg_marquardtFixed<vw::camera::CameraGenericLMA, 2,3>(model, start, objective, status,
                                               ABS_TOL, REL_TOL, MAX_ITERATIONS);
  // Check the error - If it is too high then the solver probably got stuck at the edge of the image.
  double  error = norm_2(model(solution));
  VW_ASSERT( (status > 0) && (error < MAX_ERROR),
	           vw::camera::PointToPixelErr() << "Unable to project point into LinescanSPOT model" );

  return solution;
}


void SPOTCameraModel::check_time(double time, std::string const& location) const {
  if ((time < m_min_time) || (time > m_max_time))
    vw::vw_throw(vw::ArgumentErr() << "SPOTCameraModel::"<<location
                 << ": Requested time "<<time<<" is out of bounds ("
                 << m_min_time << " <-> "<<m_max_time<<")\n");
}

vw::Vector3 SPOTCameraModel::get_camera_center_at_time(double time) const {
  check_time(time, "get_camera_center_at_time");
  return m_position_func(time);
}
vw::Vector3 SPOTCameraModel::get_camera_velocity_at_time(double time) const { 
  check_time(time, "get_camera_velocity_at_time");
  return m_velocity_func(time); 
}
vw::Quat SPOTCameraModel::get_camera_pose_at_time(double time) const {
  check_time(time, "get_camera_pose_at_time");
 return m_pose_func(time); 
}
double SPOTCameraModel::get_time_at_line(double line) const {
  if ((line < 0.0) || (static_cast<int>(line) >= m_image_size[1]))
    vw::vw_throw(vw::ArgumentErr() << "SPOTCameraModel::get_time_at_line"
                 << ": Requested line "<<line<<" is out of bounds (0"
                 << " <-> "<<m_image_size[1]<<")\n");
 return m_time_func(line); 
}



Vector3 SPOTCameraModel::get_local_pixel_vector(vw::Vector2 const& pix) const {
  
  // psi_x Is the angle from nadir line in along-track direction (lines)
  // psi_y Is the angle from nadir line in across-track direction (cols)
  // psi_x is nearly constant.  psi_y starts negative and increases with column.
  
  // Interpolate the pixel angle from the adjacent values in the lookup table.
  // - Probably should have a simple 2D interp function somewhere.  
  double     col    = pix[0];
  double min_col    = floor(col);
  double max_col    = min_col + 1.0;
  size_t min_index  = static_cast<size_t>(min_col);
  size_t max_index  = static_cast<size_t>(max_col);
  double min_weight = max_col - col;
  double max_weight = col-min_col;
  
  // Check bounds
  if ((col < 0) || (col > static_cast<double>(m_look_angles.size())-1.0))
    vw::vw_throw(vw::ArgumentErr() << "SPOTCameraModel:::get_local_pixel_vector: Requested pixel "
                 << col << " is out of bounds!\n");
  
  double psi_x = (m_look_angles[max_index].second[0]*max_weight + 
                  m_look_angles[min_index].second[0]*min_weight  );
  double psi_y = (m_look_angles[max_index].second[1]*max_weight + 
                  m_look_angles[min_index].second[1]*min_weight  );
  
  // This vector is in the SPOT5 O1 Navigation Coordinate Sytem, which 
  // differs from how we usually set up our coordinates.
  //Vector3 result = normalize(Vector3(-tan(psi_y), tan(psi_x), -1));
  
  // Convert the local vector so that it follows our usual conventions:
  //  Z down, Y flight direction, X increasing sample direction. 
  Vector3 result = normalize(Vector3(tan(psi_y), tan(psi_x), 1.0));
  //std::cout << "Pixel: " << pix << std::endl;
  //std::cout << "Col: " << col << ", min col: " << min_col << ", max col: " << max_col << std::endl;
  //std::cout << "min = " << m_look_angles[min_index].second << std::endl;
  //std::cout << "max = " << m_look_angles[max_index].second << std::endl;
  //std::cout << "Local pixel vector: " << result << std::endl;
  return result;
}

Matrix3x3 SPOTCameraModel::get_local_orbital_frame(Vector3 const& position, Vector3 const& velocity) {
  // These calculations are copied from the SPOT 123-4-58 Geometry Handbook (GAEL-P135-DOC-001)
  Vector3 Z2 = vw::math::normalize(position);
  Vector3 X2 = vw::math::normalize(vw::math::cross_prod(velocity, Z2));
  Vector3 Y2 = vw::math::cross_prod(Z2, X2);
  Matrix3x3 out;
  for (int r=0; r<3; ++r) {
    out(r,0) = X2[r];
    out(r,1) = Y2[r];
    out(r,2) = Z2[r];
  }
  return out;
}

Matrix3x3 SPOTCameraModel::get_look_rotation_matrix(double yaw, double pitch, double roll) {
/*
  // These calculations are copied from the SPOT 123-4-58 Geometry Handbook (GAEL-P135-DOC-001)
  Matrix3x3 Mp, Mr, My;
  Mp(0,0) = 1.0;         Mp(0,1) = 0.0;           Mp(0,2) = 0.0;
  Mp(1,0) = 0.0;         Mp(1,1) = cos(pitch);    Mp(1,2) = sin(pitch);
  Mp(2,0) = 0.0;         Mp(2,1) = -sin(pitch);   Mp(2,2) = cos(pitch);

  Mr(0,0) = cos(roll);   Mr(0,1) = 0.0;           Mr(0,2) = -sin(roll);
  Mr(1,0) = 0.0;         Mr(1,1) = 1.0;           Mr(1,2) = 0.0;
  Mr(2,0) = sin(roll);   Mr(2,1) = 0.0;           Mr(2,2) = cos(roll);
  
  My(0,0) = cos(yaw);    My(0,1) = -sin(yaw);     My(0,2) = 0.0;
  My(1,0) = sin(yaw);    My(1,1) = cos(yaw);      My(1,2) = 0.0;
  My(2,0) = 0.0;         My(2,1) = 0.0;           My(2,2) = 1.0; 

  Matrix3x3 out = Mp*Mr*My;
  return out;
*/  

  double cp = cos(pitch);
  double sp = sin(pitch);
  double cr = cos(roll);
  double sr = sin(roll);
  double cy = cos(yaw);
  double sy = sin(yaw);

  Matrix3x3 M;
  M(0,0) = (cr*cy);            M(0,1) = (-cr*sy);           M(0,2) = (-sr);
  M(1,0) = (cp*sy+sp*sr*cy);   M(1,1) = (cp*cy-sp*sr*sy);   M(1,2) = (sp*cr);
  M(2,0) = (-sp*sy+cp*sr*cy);  M(2,1) = (-sp*cy-cp*sr*sy);  M(2,2) = cp*cr; 
  return M;
}


/*
Notes on interpolation:

Line period = 7.5199705115e-04 = 0.000751997051

Paper recommends:
position = lagrangian interpolation --> The times happen to be spaced exactly 30 secs apart.
velocity = lagrangian interpolation
pose = linear interpolation --> The times are spaced ALMOST exactly 1.0000 seconds apart
time = Linear is only option.
*/


boost::shared_ptr<SPOTCameraModel> load_spot5_camera_model_from_xml(std::string const& path)
{

  // XYZ coordinates are in the ITRF coordinate frame which means GCC coordinates.
  // - The velocities are in the same coordinate frame, not in some local frame.

  vw_out(vw::DebugMessage,"asp") << "Loading SPOT5 camera file: " << path << std::endl;

  // Parse the SPOT5 XML file
  SpotXML xml_reader;
  xml_reader.read_xml(path);

  // Get all the initial functors
  vw::camera::LagrangianInterpolation position_func  = xml_reader.setup_position_func();
  vw::camera::LagrangianInterpolation velocity_func  = xml_reader.setup_velocity_func();
  vw::camera::LinearTimeInterpolation time_func      = xml_reader.setup_time_func();
  vw::camera::LinearPiecewisePositionInterpolation spot_pose_func = xml_reader.setup_pose_func(time_func);
  
  // The SPOT5 camera uses a different pose convention than we do, so we create
  //  a new pose interpolation functor that will return the pose in an easy to use format.
  
  // Get some information about the pose data
  double min_time      = spot_pose_func.get_t0();
  double max_time      = spot_pose_func.get_tend();
  double time_delta    = spot_pose_func.get_dt();
  size_t num_pose_vals = static_cast<size_t>(round((max_time - min_time) / time_delta));
  
  // This matrix rotates the axes of the SPOT5 model so that it is oriented with
  //  our standard linescanner coordinate frame.
  Matrix3x3 R;
  R(0,0) = -1.0; R(0,1) = 0.0; R(0,2) =  0.0;
  R(1,0) =  0.0; R(1,1) = 1.0; R(1,2) =  0.0;
  R(2,0) =  0.0; R(2,1) = 0.0; R(2,2) = -1.0;
  
  // Make a new vector of pose values in the GCC coordinate frame.
  // - This saves us from having to do all of the coordinate transforms
  //   each time a camera position is needed.
  std::vector<vw::Quat> gcc_pose(num_pose_vals);
  Vector3 position, velocity, yaw_pitch_roll;
  Matrix3x3 lo_frame, look_rotation, combined_rotation;  
  for (size_t i=0; i<num_pose_vals; ++i) {
    // Get info at this time
    double time = min_time + time_delta * static_cast<double>(i);
    position       = position_func(time);
    velocity       = velocity_func(time);
    yaw_pitch_roll = spot_pose_func(time);
    
    // TODO: There may be a small (~1 meter offset) between ITRF coordinates and WGS84 coordinates!
    
    // Get the two of rotation matrices we need
    lo_frame      = SPOTCameraModel::get_local_orbital_frame(position, velocity);
    look_rotation = SPOTCameraModel::get_look_rotation_matrix(yaw_pitch_roll[0], 
                                          yaw_pitch_roll[1], yaw_pitch_roll[2]);
    //look_rotation.set_identity(); // DEBUG assume perfect path following
    // By their powers combined these form the GCC rotation we need.
    combined_rotation = lo_frame * look_rotation*R;

    gcc_pose[i] = vw::Quat(combined_rotation);    
  }
  
  vw::camera::SLERPPoseInterpolation pose_func(gcc_pose, min_time, time_delta);

  // This is where we could set the Earth radius and mean surface elevation if we have that info.

  // Feed everything into a new camera model.
  return boost::shared_ptr<SPOTCameraModel>
    (new SPOTCameraModel(position_func, velocity_func, 
                         pose_func, time_func, 
                         xml_reader.look_angles,
                         xml_reader.image_size,
                         min_time, max_time,
                         stereo_settings().enable_correct_velocity_aberration,
                         stereo_settings().enable_correct_atmospheric_refraction));
  
} // End function load_spot_camera_model()


} // end namespace asp

