#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Camera/CameraSolve.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Camera/PleiadesXML.h>
#include <asp/Camera/LinescanPleiadesModel.h>
#include <asp/Camera/CsmModel.h>
#include <usgscsm/UsgsAstroLsSensorModel.h>

// This class implements the Pleiades linescan model
// based on the CSM model.
// TODO(oalexan1): Need to think more about the desired precision of the CSM
// model. Need high accuracy for bundle adjustment, but may get away
// with less for mapprojection. For now, err towards more accuracy.
namespace asp {
// Constructor
PleiadesCameraModel::
PleiadesCameraModel(vw::camera::LinearTimeInterpolation const& time,
                    vw::camera::LagrangianInterpolation const& position,
                    vw::camera::LagrangianInterpolation const& velocity,
                    bool isNeo, double t0Quat, double dtQuat,
                    double quat_offset_time, double quat_scale,
                    std::vector<vw::Vector<double, 4>>  const& quaternion_coeffs,
                    vw::Vector2                         const& coeff_psi_x,
                    vw::Vector2                         const& coeff_psi_y,
                    vw::Vector2i                        const& image_size,
                    double min_time, double max_time,
                    int ref_col, int ref_row, double accuracy_stdv):
  m_position_func(position), m_velocity_func(velocity),
  m_isNeo(isNeo), m_t0Quat(t0Quat), m_dtQuat(dtQuat),
  m_quat_offset_time(quat_offset_time), m_quat_scale(quat_scale),
  m_quaternion_coeffs(quaternion_coeffs),
  m_time_func(time), m_coeff_psi_x(coeff_psi_x), m_coeff_psi_y(coeff_psi_y),
  m_min_time(min_time), m_max_time(max_time), m_ref_col(ref_col), m_ref_row(ref_row),
  m_image_size(image_size), m_accuracy_stdv(accuracy_stdv) {

  populateCsmModel();
}

void PleiadesCameraModel::populateCsmModel() {

  // Populate CsmModel class members
  // Do not use a precision below 1.0-e8 as then the linescan model will return junk.
  m_desired_precision = asp::DEFAULT_CSM_DESIRED_PRECISISON;
  vw::cartography::Datum datum("WGS84"); // this sensor is used for Earth only
  m_semi_major_axis = datum.semi_major_axis();
  m_semi_minor_axis = datum.semi_minor_axis();
  
  // Create the linescan model
  m_gm_model.reset(new UsgsAstroLsSensorModel); // m_gm_model will manage the deallocation
  m_ls_model = dynamic_cast<UsgsAstroLsSensorModel*>(m_gm_model.get()); // pointer to ls model
  if (m_ls_model == NULL)
    vw::vw_throw(vw::ArgumentErr() << "Invalid initialization of the linescan model.\n");

  // This performs many initializations apart from the above
  m_ls_model->reset();

  // Override some initializations
  m_ls_model->m_nSamples = m_image_size[0]; 
  m_ls_model->m_nLines   = m_image_size[1];
  m_ls_model->m_platformFlag = 1; // explicitly set to 1, to have order 8 Lagrange interpolation
  m_ls_model->m_maxElevation =  10000.0; //  10 km
  m_ls_model->m_minElevation = -10000.0; // -10 km
  m_ls_model->m_focalLength  = 1.0;
  m_ls_model->m_zDirection   = 1.0;
  m_ls_model->m_halfSwath    = 1.0;
  m_ls_model->m_sensorIdentifier = "Pleiades";
  m_ls_model->m_majorAxis = m_semi_major_axis;
  m_ls_model->m_minorAxis = m_semi_minor_axis;
  
  m_ls_model->m_iTransL[0]   = 0.0;  
  m_ls_model->m_iTransL[1]   = 1.0; // no scale
  m_ls_model->m_iTransL[2]   = 0.0; // no skew
  m_ls_model->m_iTransS[0]   = 0.0;
  m_ls_model->m_iTransS[1]   = 0.0; // no skew
  m_ls_model->m_iTransS[2]   = 1.0; // no scale
  m_ls_model->m_detectorLineOrigin   = 0.0;
  m_ls_model->m_detectorSampleOrigin = 0.0;

  // Quantities needed to find the ray direction in the sensor plane.
  // This needs to be consistent with usgscsm functions
  // computeDistortedFocalPlaneCoordinates() and
  // createCameraLookVector(), which requires a lot of care.
  // Need to emulate this
  // double x = m_coeff_psi_x[0] + m_coeff_psi_x[1] * (col  + m_ref_col);
  // double y = m_coeff_psi_y[0] + m_coeff_psi_y[1] * (col  + m_ref_col);
  // Using this:
  // double detSample = (col + 0.5) * sampleSumming + startingSample;
  // double detLine = line * lineSumming + startingLine; // but it will use line = 0

  m_ls_model->m_detectorLineSumming    = 1.0;
  m_ls_model->m_startingDetectorLine   =  m_coeff_psi_y[0]; // note that m_coeff_psi_y[1] = 0
  m_ls_model->m_detectorSampleSumming  = -m_coeff_psi_x[1];
  m_ls_model->m_startingDetectorSample = -m_coeff_psi_x[0] - m_coeff_psi_x[1] * (m_ref_col - 0.5);

  // Time
  m_ls_model->m_intTimeLines.push_back(1.0); // to offset CSM's quirky 0.5 additions in places
  m_ls_model->m_intTimeStartTimes.push_back(m_time_func.m_t0);
  m_ls_model->m_intTimes.push_back(m_time_func.m_dt);
  int num_pos = m_position_func.m_samples.size();
  if ((size_t)num_pos != m_velocity_func.m_samples.size())
    vw::vw_throw(vw::ArgumentErr() << "Expecting as many positions as velocities.\n");

  // Positions and velocities
  m_ls_model->m_numPositions = 3 * num_pos; // concatenate all coordinates
  m_ls_model->m_t0Ephem = m_position_func.get_t0();
  
  m_ls_model->m_dtEphem = m_position_func.get_dt();
  m_ls_model->m_positions.resize(m_ls_model->m_numPositions);
  m_ls_model->m_velocities.resize(m_ls_model->m_numPositions);
  for (int pos_it = 0; pos_it < num_pos; pos_it++) {
    for (int coord = 0; coord < 3; coord++) {
      m_ls_model->m_positions [3*pos_it + coord] = m_position_func.m_samples[pos_it][coord];
      m_ls_model->m_velocities[3*pos_it + coord] = m_velocity_func.m_samples[pos_it][coord];
    }
  }

  // Quaternions. For NEO these are tabulated, while for 1A/1B they come
  // from sampling a polynomial. In the latter case, 
  // quaternions are sampled over the range of times for which the position
  // and velocity are available, which a way longer range than the time spent
  // acquiring image lines.
  // TODO(oalexan1): What is the right factor (inverse of sampling rate)?
  if (m_isNeo) {
    m_ls_model->m_numQuaternions = 4 * m_quaternion_coeffs.size();
    m_ls_model->m_t0Quat = m_t0Quat;
    m_ls_model->m_dtQuat = m_dtQuat;
  } else {
    int factor = 100;
    // quaternion t0, borrow from position t0
    m_ls_model->m_t0Quat = m_ls_model->m_t0Ephem; 
    // quaternion dt, borrow from position dt
    m_ls_model->m_dtQuat = m_ls_model->m_dtEphem / factor; 
    // Great care is needed here. Number of samples is obtained
    // by dividing the range of times by the sampling rate, and
    // then adding one. 
    double beg_time = m_ls_model->m_t0Ephem;
    double end_time = beg_time + m_ls_model->m_dtEphem * (num_pos - 1);
    int num_new_pos = (int)round((end_time - beg_time) / m_ls_model->m_dtQuat) + 1;
    m_ls_model->m_numQuaternions = 4 * num_new_pos;
  }

  m_ls_model->m_quaternions.resize(m_ls_model->m_numQuaternions);
  for (int pos_it = 0; pos_it < m_ls_model->m_numQuaternions / 4; pos_it++) {
    vw::Quat q;
    if (m_isNeo) {
      vw::Vector<double, 4> const& v = m_quaternion_coeffs[pos_it]; // alias
      q = vw::Quat(v[0], v[1], v[2], v[3]); // order is w, x, y, z
    } else {
      // Sample the polynomial
      double t = m_ls_model->m_t0Quat + pos_it * m_ls_model->m_dtQuat;
      q = get_camera_pose_at_time(t);
    }

    // ASP stores the quaternions as (w, x, y, z). CSM wants them as
    // x, y, z, w.
    int coord = 0;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.x(); coord++;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.y(); coord++;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.z(); coord++;
    m_ls_model->m_quaternions[4*pos_it + coord] = q.w(); coord++;
  }

  // Re-creating the model from the state forces some operations to
  // take place which are inaccessible otherwise.
  std::string modelState = m_ls_model->getModelState();
  m_ls_model->replaceModelState(modelState);
  
#if 0 // For debugging
  std::string json_state_file = "tmp.json";
  modelState = m_ls_model->getModelState(); // refresh this
  vw::vw_out() << "Writing model state: " << json_state_file << std::endl;
  std::ofstream ofs(json_state_file.c_str());
  ofs << modelState << std::endl;
  ofs.close();
#endif

}
  
vw::Vector3 PleiadesCameraModel::camera_center(vw::Vector2 const& pix) const {
  csm::ImageCoord csm_pix;
  asp::toCsmPixel(pix, csm_pix);

  double time = m_ls_model->getImageTime(csm_pix);
  csm::EcefCoord ecef = m_ls_model->getSensorPosition(time);

  return vw::Vector3(ecef.x, ecef.y, ecef.z);
}

vw::Vector3 PleiadesCameraModel::pixel_to_vector(vw::Vector2 const& pix) const {
  csm::ImageCoord csm_pix;
  asp::toCsmPixel(pix, csm_pix);
  
  csm::EcefLocus locus = m_ls_model->imageToRemoteImagingLocus(csm_pix);
  return vw::Vector3(locus.direction.x, locus.direction.y, locus.direction.z);
}
  
vw::Vector2 PleiadesCameraModel::point_to_pixel(vw::Vector3 const& point) const {

  csm::EcefCoord ecef(point[0], point[1], point[2]);
  
  // Do not show warnings, it becomes too verbose
  double achievedPrecision = -1.0;
  csm::WarningList warnings;
  csm::WarningList * warnings_ptr = NULL;
  bool show_warnings = false;
  csm::ImageCoord csm_pix = m_ls_model->groundToImage(ecef, m_desired_precision,
                                                      &achievedPrecision, warnings_ptr);
  
  vw::Vector2 asp_pix;
  asp::fromCsmPixel(asp_pix, csm_pix);
  return asp_pix;
}
  
// This function is tricky to implement 
vw::Quaternion<double> PleiadesCameraModel::camera_pose(vw::Vector2 const& pix) const {
  // This is not implemented for now for the CSM model
  vw_throw(vw::NoImplErr() << "LinescanPleiadesModel: Cannot retrieve camera_pose!");
  return vw::Quaternion<double>();
}
  
// Allow finding the time at any line, even negative ones. Here a
// simple slope-intercept formula is used rather than a table so one
// cannot run out of bounds. Page 76 in the doc.
double PleiadesCameraModel::get_time_at_line(double line) const {
  csm::ImageCoord csm_pix;
  asp::toCsmPixel(vw::Vector2(0, line), csm_pix);
  return m_ls_model->getImageTime(csm_pix);
}
 
// Throw an exception if the input time is outside the given
// bounds. The valid range is much bigger than the range of times
// at which image lines are recorded. It is rather the range at
// which positions, velocities, and quaternions are tabulated.
void PleiadesCameraModel::check_time(double time, std::string const& location) const {
  if ((time < m_min_time) || (time > m_max_time))
    vw::vw_throw(vw::ArgumentErr() << "PleiadesCameraModel::"<<location
                 << ": Requested time "<<time<<" is out of bounds ("
                 << m_min_time << " <-> "<<m_max_time<<")\n");
}

vw::Vector3 PleiadesCameraModel::get_camera_center_at_time(double time) const {
  // TODO(oalexan1): This needs more testing. Normally it is not invoked.
  csm::EcefCoord ecef = m_ls_model->getSensorPosition(time);
  return vw::Vector3(ecef.x, ecef.y, ecef.z);
}
  
vw::Vector3 PleiadesCameraModel::get_camera_velocity_at_time(double time) const { 
  // TODO(oalexan1): This needs testing.
  csm::EcefVector ecef = m_ls_model->getSensorVelocity(time);
  return vw::Vector3(ecef.x, ecef.y, ecef.z);
}

// Compute the quaternion at given time using the polynomial
// expression (doc page 77). This should work whether or not the CSM
// model is used. This applies only to L1A/B products, and not to the NEO
// product, when the quaternions are tabulated instead.
vw::Quat PleiadesCameraModel::get_camera_pose_at_time(double time) const {

  if (m_isNeo)
    vw::vw_throw(vw::NoImplErr() << "PleiadesCameraModel: Cannot compute camera pose "
      << "from polynomial for NEO products. This is a programmer error.\n");

  double scaled_t = (time - m_quat_offset_time) / m_quat_scale;

  if (m_quaternion_coeffs.size() != 4)
    vw::vw_throw(vw::ArgumentErr() << "Expecting four quaternions.\n");
  
  vw::Vector<double, 4> v(0, 0, 0, 0); // future quaternion
  double tn = 1.0; // scaled_t to the power of n
  for (size_t it = 0; it < 4; it++) {

    for (size_t coord = 0; coord < 4; coord++)
      v[coord] += m_quaternion_coeffs[coord][it] * tn;
    
    tn *= scaled_t;
  }
  
  return vw::Quaternion<double>(v[0], v[1], v[2], v[3]); // order is w, x, y, z
}

// The pointing vector in sensor coordinates. This function is now not used,
// because CSM does the work. 
// Page 76 in the doc.
vw::Vector3 PleiadesCameraModel::get_local_pixel_vector(vw::Vector2 const& pix) const {

  double col = pix[0];
  double row = pix[1];
  
  // The doc says to subtract m_ref_col, while it was found
  // experimentally that one needs to add it.
  double x = m_coeff_psi_x[0] + m_coeff_psi_x[1] * (col  + m_ref_col);

  // Note how below col (and not row) is used, per the Pleiades doc.
  // Note that m_coeff_psi_y[1] is 0.
  double y = m_coeff_psi_y[0] + m_coeff_psi_y[1] * (col  + m_ref_col);

  vw::Vector3 result = vw::Vector3(y, -x, 1.0);

  // Make the direction have unit length
  result = normalize(result);

  return result;
}

boost::shared_ptr<PleiadesCameraModel>
load_pleiades_camera_model_from_xml(std::string const& path) {

  // Parse the Pleiades XML file
  PleiadesXML xml_reader;
  xml_reader.read_xml(path);

  // Get all the initial functors
  vw::camera::LinearTimeInterpolation
    time_func = xml_reader.setup_time_func();
  vw::camera::LagrangianInterpolation
    position_func = xml_reader.setup_position_func(time_func);
  vw::camera::LagrangianInterpolation
    velocity_func = xml_reader.setup_velocity_func(time_func);
  
  // Set up the quaternions for NEO. This will create xml_reader m_t0Quat, 
  // m_dtQuat, m_quaternion_coeffs.
  if (xml_reader.m_isNeo)
    xml_reader.setup_pose_func(time_func);

  // Find the range of times for which we can solve for position and pose
  double min_position_time = position_func.get_t0();
  double max_position_time = position_func.get_tend();
  double min_velocity_time = velocity_func.get_t0();
  double max_velocity_time = velocity_func.get_tend();

  double min_time = std::max(min_position_time, min_velocity_time);
  double max_time = std::min(max_position_time, max_velocity_time);

  // Create the model. This can throw an exception.
  boost::shared_ptr<PleiadesCameraModel> cam
    (new PleiadesCameraModel(time_func, position_func, velocity_func,
                             xml_reader.m_isNeo,
                             xml_reader.m_t0Quat,
                             xml_reader.m_dtQuat,
                             xml_reader.m_quat_offset_time,
                             xml_reader.m_quat_scale,
                             xml_reader.m_quaternion_coeffs,
                             xml_reader.m_coeff_psi_x,
                             xml_reader.m_coeff_psi_y,
                             xml_reader.m_image_size,
                             min_time, max_time,
                             xml_reader.m_ref_col, xml_reader.m_ref_row,
                             xml_reader.m_accuracy_stdv));
  
  return cam;
} // End function load_pleiades_camera_model()

} // end namespace asp

