#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2006-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  https://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file WidgetBase.cc

#include <asp/GUI/WidgetBase.h>
#include <vw/Math/Statistics.h>

namespace vw { namespace gui {

WidgetBase::WidgetBase(int beg_image_id, int end_image_id,
             int base_image_id,
             bool use_georef,
             std::vector<imageData> & images):
    m_beg_image_id(beg_image_id),
    m_end_image_id(end_image_id),
    m_base_image_id(base_image_id), 
    m_use_georef(use_georef),
    m_images(images) {

    int num_images = m_images.size();
    m_world2image_geotransforms.resize(num_images);
    m_image2world_geotransforms.resize(num_images);
}
  
// Convert from world coordinates to projected coordinates in given geospatial
// projection
vw::Vector2 WidgetBase::world2projpoint(vw::Vector2 P, int imageIndex) const {
  if (!m_use_georef)
      return flip_in_y(P);
  return m_world2image_geotransforms[imageIndex].point_to_point(flip_in_y(P)); 
}

// The reverse of world2projpoint
vw::Vector2 WidgetBase::projpoint2world(vw::Vector2 P, int imageIndex) const {
  if (!m_use_georef)
    return flip_in_y(P);
  return flip_in_y(m_image2world_geotransforms[imageIndex].point_to_point(P));
}

// Find the min and max values, ignoring outliers. We look only 
// at the last component of each point, as that has the intensity,
// while the previous two have the position.
// Keep this here as it is used only for plotting in widget code.
void findRobustBounds(std::vector<vw::Vector3> const& scattered_data,
  double & min_val, double & max_val) {

  std::vector<double> vals;
  for (size_t pt_it = 0; pt_it < scattered_data.size(); pt_it++)
    vals.push_back(scattered_data[pt_it][2]);

  double beg_inlier = -1, end_inlier = -1, pct_fraction = 0.25, factor = 3.0;
  vw::math::find_outlier_brackets(vals, pct_fraction, factor, beg_inlier, end_inlier);
  min_val = end_inlier;
  max_val = beg_inlier;

  for (size_t it = 0; it < vals.size(); it++) {
    if (vals[it] < beg_inlier || vals[it] > end_inlier) 
      continue;
    min_val = std::min(min_val, vals[it]);
    max_val = std::max(max_val, vals[it]);
  }

  return;
}

}} // namespace vw::gui
