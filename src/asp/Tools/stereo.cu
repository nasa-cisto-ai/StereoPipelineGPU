#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file stereo.cc

#include <asp/Tools/stereo.h>
#include <asp/Camera/RPCModel.h>
#include <asp/Core/Bathymetry.h>
#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/Camera/LinescanPleiadesModel.h>
#include <asp/Core/AspStringUtils.h>

#include <vw/Cartography/PointImageManipulation.h>
#include <vw/Stereo/StereoView.h>
#include <vw/Stereo/PreFilter.h>
#include <vw/Stereo/CorrelationView.h>
#include <vw/Stereo/CostFunctions.h>
#include <vw/Stereo/DisparityMap.h>
#include <vw/FileIO/MatrixIO.h>

// Can't do much about warnings in boost except to hide them
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations"
#include <boost/accumulators/accumulators.hpp>
#include <boost/accumulators/statistics.hpp>
#pragma GCC diagnostic pop

using namespace vw;
using namespace vw::cartography;

namespace asp {

  // Transform the crop window to be in reference to L.tif
  BBox2i transformed_crop_win(ASPGlobalOptions const& opt){

    BBox2i b = stereo_settings().left_image_crop_win;
    boost::shared_ptr<vw::DiskImageResource> rsrc = 
            vw::DiskImageResourcePtr(opt.in_file1);
    DiskImageView<PixelGray<float>> left_image(rsrc);
    BBox2i full_box = bounding_box(left_image);
    if (b == BBox2i(0, 0, 0, 0)){

      // No box was provided. Use the full box.
      if ( fs::exists(opt.out_prefix+"-L.tif") ){
        DiskImageView<PixelGray<float>> L_img(opt.out_prefix+"-L.tif");
        b = bounding_box(L_img);
      }else{
        b = full_box; // To not have an empty box
      }

    }else{

      // Ensure that the region is inside the maximum theoretical region
      b.crop(full_box);

      if ( fs::exists(opt.out_prefix+"-align-L.exr") ){
        Matrix<double> align_left_matrix = math::identity_matrix<3>();
        read_matrix(align_left_matrix, opt.out_prefix + "-align-L.exr");
        b = HomographyTransform(align_left_matrix).forward_bbox(b);
      }

      if ( fs::exists(opt.out_prefix+"-L.tif") ){
        // Intersect with L.tif which is the transformed and processed left image
        DiskImageView<PixelGray<float>> L_img(opt.out_prefix+"-L.tif");
        b.crop(bounding_box(L_img));
      }

    }

    return b;
  }

  void parse_multiview(int argc, char* argv[],
                       boost::program_options::options_description const&
                       additional_options,
                       bool verbose,
                       std::string & output_prefix,
                       std::vector<ASPGlobalOptions> & opt_vec,
                       bool exit_early){

    // If a stereo program is invoked as:

    // prog <images> <cameras> <output-prefix> [<input_dem>] <other options>

    // with the number of images n >= 2, create n-1 individual
    // ASPGlobalOptions entries, corresponding to n-1 stereo pairs between the
    // first image and each of the subsequent images.

    //vw_out() << "DEBUG - parse_multiview inputs:" << std::endl;
    //for (int i=0; i<argc; ++i) vw_out() << argv[i] << std::endl;

    // First reset the outputs
    output_prefix.clear();
    opt_vec.clear();

    // Extract the images/cameras/output prefix, and perhaps the input DEM
    std::vector<std::string> files;
    bool is_multiview = true;
    ASPGlobalOptions opt;
    std::string usage;
    handle_arguments(argc, argv, opt, additional_options,
                     is_multiview, files, usage, exit_early);

    // Need this for the GUI, ensure that opt_vec is never empty, even on failures
    opt_vec.push_back(opt);

    if (files.size() < 3)
      vw_throw(ArgumentErr() << "Missing all of the correct input files.\n\n" << usage);

    // If a file shows up more than once as input, that will confuse
    // the logic at the next step, so forbid that.
    std::map<std::string, int> vals;
    for (int s = 1; s < argc; s++)
      vals[argv[s]]++;
    for (int s = 0; s < (int)files.size(); s++){
      if (vals[files[s]] > 1){
        vw_throw(ArgumentErr() << "The following input argument shows up more than "
                  << "once and hence cannot be parsed correctly: "
                  << files[s] << ".\n\n" << usage);
      }
    }

    // Store the options and their values (that is, not the input files).
    std::set<std::string> file_set;
    for (int s = 0; s < (int)files.size(); s++)
      file_set.insert(files[s]);
    std::vector<std::string> options;
    for (int s = 1; s < argc; s++){
      if (file_set.find(argv[s]) == file_set.end())
        options.push_back(argv[s]);
    }

    // Extract all the positional elements
    std::vector<std::string> images, cameras;
    std::string input_dem;
    if (!parse_multiview_cmd_files(files, images, cameras, output_prefix, input_dem))
      vw_throw(ArgumentErr() << "Missing all of the correct input files.\n\n" << usage);

    int num_pairs = (int)images.size() - 1;
    if (num_pairs <= 0)
      vw_throw(ArgumentErr() << "Insufficient number of images provided.\n");

    if (num_pairs > 1 && stereo_settings().propagate_errors) 
      vw::vw_throw(vw::ArgumentErr() << "Error propagation is not "
                   << "implemented for more than two images.\n");
 
    // Must signal to the children runs that they are part of a multiview run
    if (num_pairs > 1) {
      std::string opt_str = "--part-of-multiview-run";
      std::vector<std::string>::iterator it = find(options.begin(), options.end(),
                                                   opt_str);
      if (it == options.end())
        options.push_back(opt_str);
    }

    // Multiview is very picky about alignment method
    if ( (num_pairs > 1 || stereo_settings().part_of_multiview_run) &&
         stereo_settings().alignment_method != "none"               &&
         stereo_settings().alignment_method != "homography"){

      std::string new_alignment;
      if (input_dem == "")
        new_alignment = "homography";
      else
        new_alignment = "none";

      vw_out(WarningMessage)
        << "For multi-view stereo, only alignment method of 'none' or 'homography' "
        << "is supported. Changing alignment method from '"
        << stereo_settings().alignment_method << "' to '" << new_alignment << "'.\n";
      stereo_settings().alignment_method = new_alignment;

      // Set this for future runs as well
      std::string align_opt = "--alignment-method";
      auto it = std::find(options.begin(), options.end(), align_opt);
      if (it != options.end() && it + 1 != options.end()){
        // Modify existing alignment
        *(it+1) = new_alignment;
      }else{
        // Set new alignment
        options.push_back(align_opt);
        options.push_back(new_alignment);
      }
     }

    // Needed for stereo_parse
    if (verbose)
      vw_out() << "num_stereo_pairs," << num_pairs << std::endl;

    std::string prog_name = extract_prog_name(argv[0]);

    // Generate the stereo command for each of the pairs made up of the first
    // image and each subsequent image, with corresponding cameras.
    opt_vec.resize(num_pairs);
    for (int p = 1; p <= num_pairs; p++){

      std::vector<std::string> cmd;
      cmd.push_back(prog_name);

      // The command line options go first
      for (int t = 0; t < (int)options.size(); t++)
        cmd.push_back(options[t]);

      cmd.push_back(images[0]); // left image
      cmd.push_back(images[p]); // right image

      if (!cameras.empty()){
        // Don't append the camera names if they are the same as the image names
        if ((images[0] != cameras[0]) && (images[p] != cameras[p])){
          cmd.push_back(cameras[0]); // left camera
          cmd.push_back(cameras[p]); // right camera
        }
      }

      std::string local_prefix = output_prefix;
      if (num_pairs > 1){
        // Need to have a separate output prefix for each pair
        std::ostringstream os;
        os << local_prefix << "-pair" << p << "/" << p;
        local_prefix = os.str();
      }
      cmd.push_back(local_prefix);

      if (!input_dem.empty())
        cmd.push_back(input_dem);

      // Create a local argc and argv for the given stereo pair and parse them.
      int largc = cmd.size();
      std::vector<char*> largv;
      for (int t = 0; t < largc; t++)
        largv.push_back((char*)cmd[t].c_str());
      ASPGlobalOptions opt;
      bool is_multiview = false;
      std::vector<std::string> files;
      handle_arguments(largc, &largv[0], opt, additional_options,
                       is_multiview, files, usage, exit_early);
      opt_vec[p-1] = opt;

      if (verbose) {
        // Needed for stereo_parse
        int big = 10000; // helps keep things in order in the python script
        vw_out() << "multiview_command_" << big + p << ",";
        for (int t = 0; t < largc-1; t++)
          vw_out() << cmd[t] << ",";
        if (largc > 0)
          vw_out() << cmd[largc-1];
        vw_out() << std::endl;
      }

    }

    if (num_pairs > 1 && prog_name != "stereo_parse" &&
        prog_name != "stereo_tri" && prog_name != "stereo_gui")
      vw_throw(ArgumentErr() << "The executable " << prog_name
               << " is not meant to be used directly with more than two images. "
               << "Use instead the stereo/parallel_stereo scripts with desired entry points.\n");

    // This must not happen earlier as StereoSession is not initialized yet
    if (num_pairs > 1 && opt.session->do_bathymetry()) 
      vw_throw(ArgumentErr() << "Bathymetry correction does not work with "
               << "multiview stereo.\n");
    
    if (opt.session->do_bathymetry() && stereo_settings().propagate_errors) 
      vw_throw(ArgumentErr() << "Error propagation is not implemented when "
               << "bathymetry is modeled.\n");
    
    if (stereo_settings().propagate_errors && 
        stereo_settings().compute_error_vector) 
      vw::vw_throw(vw::ArgumentErr() << "Cannot use option --error-vector for computing "
                   << "the triangulation error vector when propagating errors (covariances) "
                   << "from cameras, as those are stored instead in " 
                   << "bands 5 and 6.\n");
    
    return;
  }

  // Parse data needed for error propagation
  void setup_error_propagation(ASPGlobalOptions const& opt) {
    
    // A bugfix for the propagated errors not being saved with enough digits
    if (stereo_settings().point_cloud_rounding_error > 0) {
      vw_out(WarningMessage) << "Option --point-cloud-rounding-error is set to " <<
        stereo_settings().point_cloud_rounding_error << " meters. If too coarse, "
        "it may create artifacts in the propagated horizontal and vertical errors.\n";
    } else {
        stereo_settings().point_cloud_rounding_error = 1.0e-8;
        vw_out() << "Round triangulated points to "
                 << stereo_settings().point_cloud_rounding_error << " meters. "
                 << "(Option: --point-cloud-rounding-error.) "  
                 << "This is much finer rounding than usual, motivated by the "
                 << "fact that the propagated errors vary slowly and will be "
                 << "saved with step artifacts otherwise.\n";
    }
    
    vw::Vector2 & v = asp::stereo_settings().horizontal_stddev; // alias, will modify
    
    if (v[0] < 0 || v[1] < 0) 
      vw::vw_throw(vw::ArgumentErr() << "Cannot have negative horizontal stddev.\n");
    
    if (int(v[0] > 0) + int(v[1] > 0) == 1)
      vw::vw_throw(vw::ArgumentErr() << "Cannot have one positive horizontal "
                   << "stddev and the other one equal to zero.\n");

    if (v[0] == 0 && v[1] == 0) {

      // This will not reload the cameras
      boost::shared_ptr<camera::CameraModel> camera_model1, camera_model2;
      opt.session->camera_models(camera_model1, camera_model2);

      // Try to create horizontal stddev based on the RPC camera files
      const asp::RPCModel *rpc1
        = dynamic_cast<const asp::RPCModel*>(vw::camera::unadjusted_model(camera_model1.get()));
      const asp::RPCModel *rpc2
        = dynamic_cast<const asp::RPCModel*>(vw::camera::unadjusted_model(camera_model2.get()));
      if (rpc1 != NULL && rpc2 != NULL) {
        double bias1 = rpc1->m_err_bias, rand1 = rpc1->m_err_rand;
        double bias2 = rpc2->m_err_bias, rand2 = rpc2->m_err_rand;
        if (bias1 > 0 && rand1 > 0 && bias2 > 0 && rand2 > 0) {
          vw_out() << "Computing horizontal stddev values from RPC camera files.\n";
          v.x() = sqrt(bias1 * bias1 + rand1 * rand1);
          v.y() = sqrt(bias2 * bias2 + rand2 * rand2);
        } else {
          vw::vw_throw(vw::ArgumentErr()
                       << "Cannot propagate errors, as no input horizontal "
                       << "stddev values were specified, and the RPC camera models lacks "
                       << "the necessary ERRBIAS and ERRRAND fields that could be used "
                       << "instead.\n");
        }
      }

      // Try to create horizontal stddev based on the Pleiades camera files
      const asp::PleiadesCameraModel *pleiades1
        = dynamic_cast<const asp::PleiadesCameraModel*>
        (vw::camera::unadjusted_model(camera_model1.get()));
      const asp::PleiadesCameraModel *pleiades2
        = dynamic_cast<const asp::PleiadesCameraModel*>
        (vw::camera::unadjusted_model(camera_model2.get()));
      if (pleiades1 != NULL && pleiades2 != NULL) {
        double accuracy1 = pleiades1->m_accuracy_stdv;
        double accuracy2 = pleiades2->m_accuracy_stdv;
        if (accuracy1 > 0 && accuracy2 > 0) {
          vw_out() << "Reading horizontal stddev values from Pleiades linescan camera files.\n";
          v.x() = accuracy1; 
          v.y() = accuracy2;
        } else {
          vw::vw_throw(vw::ArgumentErr()
                       << "Cannot propagate errors, as no input horizontal "
                       << "stddev values were specified, and the Pleiades camera models lacks "
                       << "the necessary ACCURACY_STDV field that could be used "
                       << "instead.\n");
        }
      }
    }
    
    bool isDg = (opt.session->name() == "dg" || opt.session->name() == "dgmaprpc");
    if (v[0] > 0 && v[1] > 0) {
      vw_out() << "Horizontal stddev per camera: " << v[0]  << ", " << v[1] 
      << " (meters).\n";
      if (isDg) 
        vw_out() << "Will use these to find the point cloud stddev, rather than "
                 << "satellite position and orientation covariances.\n";
    } else {
      if (!isDg)
        vw::vw_throw(vw::ArgumentErr()
                     << "Cannot propagate errors, as no input horizontal "
                     << "stddev values were specified.\n");
    }
      
  }
  
  // Parse input command line arguments
  void handle_arguments(int argc, char *argv[], ASPGlobalOptions& opt,
                        boost::program_options::options_description const&
                        additional_options,
                        bool is_multiview, std::vector<std::string> & input_files,
                        std::string & usage, bool exit_early){

    // Add options whose values are stored in ASPGlobalOptions rather than in stereo_settings()
    po::options_description general_options_sub("");
    addAspGlobalOptions(general_options_sub, opt);

    // We distinguish between all_general_options, which is all the
    // options we must parse, even if we don't need some of them, and
    // general_options, which are the options specifically used by the
    // current tool, and for which we also print the help message.

    po::options_description general_options("");
    general_options.add(general_options_sub);
    general_options.add(additional_options);
    general_options.add(vw::GdalWriteOptionsDescription(opt));

    po::options_description all_general_options("");
    all_general_options.add(general_options_sub );
    all_general_options.add(generate_config_file_options(opt));

    po::options_description positional_options("");
    po::positional_options_description positional_desc;
    if (is_multiview){
      // The number of input files could be huge. Just store them in a vector,
      // we'll parse them in the caller.
      positional_options.add_options()
        ("input-files", po::value< std::vector<std::string>>(), "Input files");
      positional_desc.add("input-files", -1);
    }else{
      // Two-view, have left and right.
      positional_options.add_options()
        ("left-input-image",   po::value(&opt.in_file1),   "Left input image")
        ("right-input-image",  po::value(&opt.in_file2),   "Right input image")
        ("left-camera-model",  po::value(&opt.cam_file1),  "Left camera model file")
        ("right-camera-model", po::value(&opt.cam_file2),  "Right camera model file")
        ("output-prefix",      po::value(&opt.out_prefix), "Prefix for output filenames")
        ("input-dem",          po::value(&opt.input_dem),  "Input DEM");

      positional_desc.add("left-input-image",   1);
      positional_desc.add("right-input-image",  1);
      positional_desc.add("left-camera-model",  1);
      positional_desc.add("right-camera-model", 1);
      positional_desc.add("output-prefix",      1);
      positional_desc.add("input-dem",          1);
    }

    usage = "[options] <images> [<cameras>] <output_file_prefix> [DEM]\n  Extensions are automatically added to the output files.\n  Camera model arguments may be optional for some stereo session types (e.g., isis).\n  Stereo parameters should be set in the stereo.default file.";
    bool allow_unregistered = false;
    std::vector<std::string> unregistered;
    po::variables_map vm = asp::check_command_line(argc, argv, opt, general_options,
                                                   all_general_options, positional_options,
                                                   positional_desc, usage,
                                                   allow_unregistered, unregistered);

    // Read the config file
    try {
      // The user can specify the positional input from the
      // stereo.default if they want to.
      po::options_description cfg_options;
      cfg_options.add(positional_options);
      cfg_options.add(generate_config_file_options(opt));

      // Append the options from the config file. Do not overwrite the
      // options already set on the command line.
      bool print_warnings = is_multiview; // print warnings just first time
      po::store(parse_asp_config_file(print_warnings,
                                      opt.stereo_default_filename,
                                      cfg_options), vm);
      po::notify(vm);
    } catch (po::error const& e) {
      vw::vw_throw(vw::ArgumentErr() << "Error parsing configuration file:\n" << e.what() << "\n");
    }

    asp::stereo_settings().validate();

    // Do this early, before any cameras are loaded
    bool print_dg_csm_cov_message = false;
    if (stereo_settings().propagate_errors) {
      if (!stereo_settings().dg_use_csm) {
        vw::Vector2 const& v = asp::stereo_settings().horizontal_stddev; // alias
        if (v[0] <= 0 || v[1] <= 0) {
          // Have to use the CSM model to propagate the errors
          // Will print a message later, only when we know the camera is actually DG
          stereo_settings().dg_use_csm = true;
          print_dg_csm_cov_message = true;
        }
      }
    }
        
    if (stereo_settings().correlator_mode) {
      stereo_settings().alignment_method = "none"; // images are assumed aligned
      opt.stereo_session = "rpc";                  // since inputs are images this seems simpler

      if (stereo_settings().propagate_errors)
        vw::vw_throw(vw::ArgumentErr() << "Cannot propagate errors in correlator mode.\n");
    }

    // Make sure that algorithm 0 is same as asp_bm, etc.
    boost::to_lower(stereo_settings().stereo_algorithm);
    if (stereo_settings().stereo_algorithm == "0") 
      stereo_settings().stereo_algorithm = "asp_bm";
    else if (stereo_settings().stereo_algorithm == "1") 
      stereo_settings().stereo_algorithm = "asp_sgm";
    else if (stereo_settings().stereo_algorithm == "2") 
      stereo_settings().stereo_algorithm = "asp_mgm";
    else if (stereo_settings().stereo_algorithm == "3") 
      stereo_settings().stereo_algorithm = "asp_final_mgm";
    
    // Add the options to the usage
    std::ostringstream os;
    os << usage << general_options;
    usage = os.str();

    // For multiview, just store the files and return
    if (is_multiview){
      if (vm.count("input-files") == 0)
        vw_throw(ArgumentErr() << "Missing input arguments.\n" << usage);
      input_files = vm["input-files"].as< std::vector<std::string>>();
      return;
    }

    // Re-use the logic in parse_multiview_cmd_files, but just for two images/cameras.
    std::vector<std::string> files;
    std::vector<std::string> images, cameras;
    if (!opt.in_file1.empty())   files.push_back(opt.in_file1);
    if (!opt.in_file2.empty())   files.push_back(opt.in_file2);
    if (!opt.cam_file1.empty())  files.push_back(opt.cam_file1);
    if (!opt.cam_file2.empty())  files.push_back(opt.cam_file2);
    if (!opt.out_prefix.empty()) files.push_back(opt.out_prefix);
    if (!opt.input_dem.empty())  files.push_back(opt.input_dem);
    if (!parse_multiview_cmd_files(files, // inputs
                                   images, cameras, opt.out_prefix, opt.input_dem)) // outputs
      vw_throw(ArgumentErr() << "Missing all of the correct input files.\n\n" << usage);

    opt.in_file1 = "";  if (images.size() >= 1)  opt.in_file1  = images[0];
    opt.in_file2 = "";  if (images.size() >= 2)  opt.in_file2  = images[1];
    opt.cam_file1 = ""; if (cameras.size() >= 1) opt.cam_file1 = cameras[0];
    opt.cam_file2 = ""; if (cameras.size() >= 2) opt.cam_file2 = cameras[1];

    if (opt.in_file1.empty() || opt.in_file2.empty() || opt.out_prefix.empty())
      vw_throw(ArgumentErr() << "Missing all of the correct input files.\n\n" << usage);

    // Create the output directory
    vw::create_out_dir(opt.out_prefix);

    // Turn on logging to file, except for stereo_parse, as that one is called
    // all the time.
    std::string prog_name = extract_prog_name(argv[0]);
    if (prog_name.find("stereo_parse") == std::string::npos) 
      asp::log_to_file(argc, argv, opt.stereo_default_filename, opt.out_prefix);
    
    // There are two crop win boxes, in respect to original left
    // image, named left_image_crop_win, and in respect to the
    // transformed left image (L.tif), named trans_crop_win. We use
    // the second if available, otherwise we transform and use the
    // first. The box trans_crop_win is for internal use, invoked
    // from parallel_stereo.

    // Interpret the the last two coordinates of the crop win boxes as
    // width and height rather than max_x and max_y. 
    BBox2i bl = stereo_settings().left_image_crop_win;
    BBox2i br = stereo_settings().right_image_crop_win;
    BBox2i bt = stereo_settings().trans_crop_win;
    stereo_settings().left_image_crop_win
      = BBox2i(bl.min().x(), bl.min().y(), bl.max().x(), bl.max().y());
    stereo_settings().right_image_crop_win
      = BBox2i(br.min().x(), br.min().y(), br.max().x(), br.max().y());
    stereo_settings().trans_crop_win
      = BBox2i(bt.min().x(), bt.min().y(), bt.max().x(), bt.max().y());

    // Ensure the crop windows are always contained in the images.
    boost::shared_ptr<vw::DiskImageResource> left_resource, right_resource;
    left_resource  = vw::DiskImageResourcePtr(opt.in_file1);
    right_resource = vw::DiskImageResourcePtr(opt.in_file2);
    DiskImageView<float> left_image(left_resource);
    DiskImageView<float> right_image(right_resource);
    stereo_settings().left_image_crop_win.crop (bounding_box(left_image));
    stereo_settings().right_image_crop_win.crop(bounding_box(right_image));

    bool crop_left  = (stereo_settings().left_image_crop_win  != BBox2i(0, 0, 0, 0));
    bool crop_right = (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0));

    // If crops were specified, check that they are valid.
    if (crop_left && stereo_settings().left_image_crop_win.empty())
      vw_throw(ArgumentErr() << "Invalid left crop window specified!\n");
    if (crop_right && stereo_settings().right_image_crop_win.empty())
      vw_throw(ArgumentErr() << "Invalid right crop window specified!\n");
    
    // Make sure the trans_crop_win value is correct going forwards.
    if (!crop_left) {
      // The crop window after transforming the left image via
      // affine epipolar or homography alignment.
      if (stereo_settings().trans_crop_win == BBox2i(0, 0, 0, 0))
        stereo_settings().trans_crop_win = transformed_crop_win(opt);

      // Intersect with L.tif which is the transformed and processed left image.
      if ( fs::exists(opt.out_prefix+"-L.tif") ){
        DiskImageView<PixelGray<float>> L_img(opt.out_prefix+"-L.tif");
        stereo_settings().trans_crop_win.crop(bounding_box(L_img));
      }
    } else {
      // If left_image_crop_win is specified, as can be see in
      // StereoSession::preprocessing_hook(), we actually
      // physically crop the image.  The trans_crop_win as passed 
      // here from parallel_stereo will already be a tile in the 
      // cropped image. So we just use it as it is. If it is not defined, 
      // we set it to the entire cropped image.
      if (stereo_settings().trans_crop_win == BBox2i(0, 0, 0, 0)) {
        stereo_settings().trans_crop_win = bounding_box(left_image);
        if ( fs::exists(opt.out_prefix+"-L.tif") ){
          DiskImageView<PixelGray<float>> L_img(opt.out_prefix+"-L.tif");
          stereo_settings().trans_crop_win = bounding_box(L_img);
        }
      }
    } // End crop checking case

    // If not using crop wins but the crop image exists, then things won't go well.
    if (!crop_left && !crop_right &&
        (fs::exists(opt.out_prefix+"-L-cropped.tif") ||
         fs::exists(opt.out_prefix+"-R-cropped.tif"))) 
      vw_throw(ArgumentErr() << "The current output prefix '" << opt.out_prefix
               << "' has an old run which used --left-image-crop-win, "
               << "but the current run does not. Results will be incorrect. "
               << "Use a new output prefix.");
    
    // TODO: May need to update this check for individual crop cases.
    // Sanity check. Don't run it if we have L-cropped.tif or R-cropped.tif,
    // in that case we have ran the gui before, and the sizes of the subimages
    // could be anything. We'll regenerate any of those anyway soon.
    if ((stereo_settings().trans_crop_win.width () <= 0 ||
         stereo_settings().trans_crop_win.height() <= 0) &&
        !fs::exists(opt.out_prefix+"-L-cropped.tif")     &&
        !fs::exists(opt.out_prefix+"-R-cropped.tif") ){
      vw_throw(ArgumentErr() << "Invalid region for doing stereo.\n\n"
               << usage << general_options);
    }

    // Ensure good order
    BBox2 & b = stereo_settings().lon_lat_limit; // alias
    if (b != BBox2(0,0,0,0)) {
      if (b.min().y() > b.max().y()) 
	      std::swap(b.min().y(), b.max().y());
      if (b.min().x() > b.max().x()) 
	      std::swap(b.min().x(), b.max().x());
    }

    if (!stereo_settings().match_files_prefix.empty() &&
        !stereo_settings().clean_match_files_prefix.empty()) 
      vw_throw(ArgumentErr() << "Cannot specify both --match-files-prefix and "
               << "--clean-match-files-prefix.\n\n" << usage << general_options);
    
    if (!stereo_settings().corr_search_limit.empty() && stereo_settings().max_disp_spread > 0)
      vw_throw(ArgumentErr() << "Cannot specify both --corr-search-limit and "
               << "--max-disp-spread.\n\n" << usage << general_options);

    // Verify that there is only one channel per input image
    if ( (left_resource->channels() > 1) || (right_resource->channels() > 1) )
      vw_throw(ArgumentErr() << "Error: Input images can only have a single channel.\n\n"
               << usage << general_options);

    if ((stereo_settings().bundle_adjust_prefix != "") &&
        (stereo_settings().alignment_method == "epipolar"))
      vw_throw(ArgumentErr() << "Error: Epipolar alignment does not support using a "
               << "bundle adjust prefix.\n\n" << usage << general_options);
    
    // Replace normal default values with these when SGM is enabled.
    // - TODO: Move these somewhere easier to find!
    const int SGM_DEFAULT_SUBPIXEL_MODE        = 12; // Blend
    const int SGM_DEFAULT_COST_MODE            = 4;
    const int SGM_DEFAULT_KERNELSIZE           = 5;
    const int SGM_DEFAULT_RM_CLEANUP_PASSES    = 0;
    const int SGM_DEFAULT_MEDIAN_FILTER_SIZE   = 3;
    const int SGM_DEFAULT_TEXTURE_SMOOTH_SIZE  = 11;
    const double SGM_DEFAULT_TEXTURE_SMOOTH_SCALE = 0.13;

    // Increase the timeout for MGM, unless specified, as that one is slow.
    // Need some care here, to make sure that even if the parent function is called
    // twice, the increase happens just once.
    if (stereo_settings().stereo_algorithm == "mgm" &&
        stereo_settings().corr_timeout == stereo_settings().default_corr_timeout) {
        stereo_settings().corr_timeout = 10 * stereo_settings().default_corr_timeout;
      vw_out() << "For the original mgm algorithm increasing the --corr-timeout to: " <<
        stereo_settings().corr_timeout << ".\n";
    }

    if (stereo_settings().correlator_mode && !opt.input_dem.empty())
      vw_throw(ArgumentErr() << "Error: With --correlator-mode, use only two "
               << "input images and no reference DEM.\n");
      
    // TODO: Modify SGM tile sizes?
    
    vw::stereo::CorrelationAlgorithm stereo_alg
      = asp::stereo_alg_to_num(stereo_settings().stereo_algorithm);
    
    // This is for asp_sgm, asp_mgm, asp_final_mgm    
    bool using_sgm = (stereo_alg > vw::stereo::VW_CORRELATION_BM && 
        stereo_alg < vw::stereo::VW_CORRELATION_OTHER);

    if (stereo_alg > vw::stereo::VW_CORRELATION_BM) {
      // If these parameters were not specified by the user, override
      // the normal default values.  Note that by setting
      // subpixel_mode to SGM_DEFAULT_SUBPIXEL_MODE, we will do no
      // further subpixel refinement than what all algorithms except
      // ASP's block matching are intrinsically capable of.  if the
      // user however explicitly specifies, for example,
      // --subpixel-mode 3, that one will be used later on.

      // These are also useful with external algorithms, as the results are then
      // smoother.
      if (vm["rm-cleanup-passes"].defaulted())
        stereo_settings().rm_cleanup_passes = SGM_DEFAULT_RM_CLEANUP_PASSES;
      if (vm["median-filter-size"].defaulted())
        stereo_settings().median_filter_size = SGM_DEFAULT_MEDIAN_FILTER_SIZE;
      if (vm["texture-smooth-size"].defaulted())
        stereo_settings().disp_smooth_size = SGM_DEFAULT_TEXTURE_SMOOTH_SIZE;
      if (vm["texture-smooth-scale"].defaulted())
        stereo_settings().disp_smooth_texture = SGM_DEFAULT_TEXTURE_SMOOTH_SCALE;
      if (vm["subpixel-mode"].defaulted()) 
        stereo_settings().subpixel_mode = SGM_DEFAULT_SUBPIXEL_MODE;
      else
        vw_out() << "Will refine the disparity using the ASP subpixel-mode: "
                 << stereo_settings().subpixel_mode << ".\n";

      // This is for the case when settings are read from stereo.default. Print
      // some warnings.
      if (stereo_settings().rm_cleanup_passes != SGM_DEFAULT_RM_CLEANUP_PASSES)
        vw_out(WarningMessage) << "When using a stereo algorithm rather than asp_bm, "
                               << "the default suggested value for "
                               << "rm-cleanup-passes is "
                               << SGM_DEFAULT_RM_CLEANUP_PASSES << ". "
                               << "Got instead " << stereo_settings().rm_cleanup_passes 
                               << ".\n";
      if (stereo_settings().median_filter_size != SGM_DEFAULT_MEDIAN_FILTER_SIZE)
        vw_out(WarningMessage) << "When using a stereo algorithm rather than asp_bm, "
                               << "the default suggested value for "
                               << "median-filter-size is "
                               << SGM_DEFAULT_MEDIAN_FILTER_SIZE << ". "
                               << "Got instead " << stereo_settings().median_filter_size
                               << ".\n";
      if (stereo_settings().disp_smooth_size != SGM_DEFAULT_TEXTURE_SMOOTH_SIZE)
        vw_out(WarningMessage) << "When using a stereo algorithm rather than asp_bm, "
                               << "the default suggested value for "
                               << "texture-smooth-size is "
                               << SGM_DEFAULT_TEXTURE_SMOOTH_SIZE << ". "
                               << "Got instead " << stereo_settings().disp_smooth_size
                               << ".\n";
      if (stereo_settings().disp_smooth_texture != SGM_DEFAULT_TEXTURE_SMOOTH_SCALE)
        vw_out(WarningMessage) << "When using a stereo algorithm rather than asp_bm, "
                               << "the default suggested value for "
                               << "texture-smooth-scale is "
                               << SGM_DEFAULT_TEXTURE_SMOOTH_SCALE << ". "
                               << "Got instead " << stereo_settings().disp_smooth_texture
                               << ".\n";
    }

    // Settings specifically for asp_sgm, asp_mgm, asp_final_mgm
    if (using_sgm) {

      if (vm["corr-kernel"].defaulted())
        stereo_settings().corr_kernel = Vector2i(SGM_DEFAULT_KERNELSIZE, SGM_DEFAULT_KERNELSIZE);

      if (vm["cost-mode"].defaulted())
        stereo_settings().cost_mode = SGM_DEFAULT_COST_MODE;

      // This is a fix for the user setting cost-mode in stereo.default, when
      // it is not defaulted. Do not allow cost mode to be different than
      // 3 or 4 for asp_sgm / asp_mgm, as it produced junk.
      if (stereo_settings().cost_mode != 3 && stereo_settings().cost_mode != 4)
        vw_throw(ArgumentErr() << "When using the asp_sgm or asp_mgm "
                << "stereo algorithm, cost-mode must be 3 or 4.\n");

      // Also do not allow corr-kernel to be outside of [3, 9]
      if (stereo_settings().corr_kernel[0] < 3 || stereo_settings().corr_kernel[0] > 9) 
        vw_throw(ArgumentErr() << "For the asp_sgm / asp_mgm algorithm, "
          << "the corr kernel size must be between 3 and 9 (inclusive).\n");
    }

    bool using_tiles = (stereo_alg > vw::stereo::VW_CORRELATION_BM ||
                        stereo_settings().alignment_method == "local_epipolar");
    if (!using_tiles) {
      // No need for a collar when we are not using tiles.
      stereo_settings().sgm_collar_size = 0;
    } 

    if (stereo_alg >= vw::stereo::VW_CORRELATION_OTHER &&
        stereo_settings().alignment_method != "local_epipolar") {
      vw_throw(ArgumentErr() << "External stereo algorithms can be "
               << "used only with alignment method local_epipolar.\n");
    }

    if (exit_early) 
      return;
    
    // The StereoSession call automatically determines the type of
    // object to create from the input parameters.
    opt.session.reset(asp::StereoSessionFactory::create(opt.stereo_session, // can change
                                                        opt,
                                                        opt.in_file1,   opt.in_file2,
                                                        opt.cam_file1,  opt.cam_file2,
                                                        opt.out_prefix, opt.input_dem));

    // Load the cameras. They will be cached in the session.
    boost::shared_ptr<camera::CameraModel> camera_model1, camera_model2;
    opt.session->camera_models(camera_model1, camera_model2);
    
    // The printing of this message was delayed till we knew the session
    if (print_dg_csm_cov_message &&
        (opt.session->name() == "dg" || opt.session->name() == "dgmaprpc"))
      vw_out() << "Enabling option --dg-use-csm as error propagation will take place. "
               << "No velocity aberration or atmospheric correction happens.\n";

    // Run a set of checks to make sure the settings are compatible.
    user_safety_checks(opt);

    // This logic must happen after the cameras are loaded.
    if (stereo_settings().propagate_errors)
      setup_error_propagation(opt);

    // The last thing we do before we get started is to copy the
    // stereo.default settings over into the results directory so that
    // we have a record of the most recent stereo.default that was used
    // with this data set.
    asp::stereo_settings().write_copy(argc, argv,
                                      opt.stereo_default_filename,
                                      opt.out_prefix + "-stereo.default");
  }

  // Register Session types
  void stereo_register_sessions() {

#if defined(ASP_HAVE_PKG_ISISIO) && ASP_HAVE_PKG_ISISIO == 1
    // Register the Isis file handler with the Vision Workbench DiskImageResource system.
    DiskImageResource::register_file_type(".cub",
                                          DiskImageResourceIsis::type_static(),
                                          &DiskImageResourceIsis::construct_open,
                                          &DiskImageResourceIsis::construct_create);
#endif
  }

  void user_safety_checks(ASPGlobalOptions const& opt){

    // Error checking

    const bool dem_provided = !opt.input_dem.empty();

    // Seed mode valid values
    if (stereo_settings().seed_mode > 3){
      vw_throw(ArgumentErr() << "Invalid value for seed-mode: "
               << stereo_settings().seed_mode << ".\n");
    }

    // D_sub from DEM needs a positive disparity_estimation_dem_error
    if (stereo_settings().seed_mode == 2 &&
        stereo_settings().disparity_estimation_dem_error <= 0.0){
      vw_throw(ArgumentErr() << "For seed-mode 2, the value of disparity-estimation-dem-error "
               << "must be positive.");
    }

    // D_sub from DEM needs a DEM
    if (stereo_settings().seed_mode == 2 &&
        stereo_settings().disparity_estimation_dem.empty()){
      vw_throw(ArgumentErr() << "For seed-mode 2, an input DEM must be provided.\n");
    }

    // D_sub from DEM does not work with map-projected images
    if (dem_provided && stereo_settings().seed_mode == 2)
      vw_throw(NoImplErr() << "Computation of low-resolution disparity from "
                << "DEM is not implemented for map-projected images.\n");

    // Must use map-projected images if input DEM is provided
    GeoReference georef1, georef2;
    bool has_georef1 = vw::cartography::read_georeference(georef1, opt.in_file1);
    bool has_georef2 = vw::cartography::read_georeference(georef2, opt.in_file2);
    if (dem_provided && (!has_georef1 || !has_georef2)){
      vw_throw(ArgumentErr() << "The images are not map-projected, "
                << "cannot use the provided DEM: " << opt.input_dem << "\n");
    }

    // If the images are map-projected, they need to use the same projection.
    if (dem_provided &&
        georef1.overall_proj4_str() != georef2.overall_proj4_str()){
      vw_throw(ArgumentErr() << "The left and right images must use the same projection.\n");
    }

    //TODO: Clean up these conditional using some kind of enum system

    // If the images are map-projected, and the cameras are specified
    // separately from the images, we need an input DEM, as we use the
    // ASP flow with map-projected images.
    if (has_georef1 && has_georef2 && !dem_provided &&
        (opt.cam_file1 != opt.in_file1) && (opt.cam_file2 != opt.in_file2) &&
        !opt.cam_file1.empty() && !opt.cam_file2.empty() ) {
        
      vw_out() << "Georef 1: " << georef1 << std::endl;
      vw_out() << "Georef 2: " << georef1 << std::endl;
      
      vw_out(WarningMessage) << "It appears that the input images are "
                             << "map-projected. In that case a DEM needs to be "
                             << "provided for stereo to give correct results.\n";
    }

    // Check that if the user provided a dem that we are using a map projection method
    if (dem_provided && !opt.session->uses_map_projected_inputs()) {
      vw_throw(ArgumentErr() << "Cannot use map-projected images with a session of type: "
                             << opt.session->name() << ".\n");
    }

    // No alignment must be set for map-projected images.
    if (stereo_settings().alignment_method != "none" && dem_provided) {
        stereo_settings().alignment_method  = "none";
      vw_out(WarningMessage) << "Changing the alignment method to 'none' "
                             << "as the images are map-projected." << std::endl;
    }

    if (dem_provided) {

      // Given session XmapY make sure that the mapprojected images were
      // done with camera Y. Normally X equals Y, with the exceptions
      // of dgmaprpc, spot5maprpc, and astermaprpc.
      std::string cam_tag = "CAMERA_MODEL_TYPE";
      std::string l_cam_type = vw::cartography::read_header_string(opt.in_file1, cam_tag);
      std::string r_cam_type = vw::cartography::read_header_string(opt.in_file2, cam_tag);

      // Extract the 'rpc' from 'rpcmaprpc' and 'dgmaprc', and 'pinhole' from 'pinholemappinhole'
      std::string tri_cam_type, mapproj_cam_type; 
      asp::parseCamTypes(opt.session->name(), tri_cam_type, mapproj_cam_type);

      // Sanity check. note that l_cam_type and r_cam_type can be empty
      if ((l_cam_type != "" && l_cam_type != mapproj_cam_type) ||
          (r_cam_type != "" && r_cam_type != mapproj_cam_type)   ){
        vw_throw(ArgumentErr() << "For session type "
                 << opt.session->name()
                 << ", the images should have been map-projected with "
                 << "the option -t \"" << mapproj_cam_type << "\". Instead, got: \""
                 << l_cam_type << "\" and \"" << r_cam_type << "\".\n");
      }

      // Can't use match files prefix with mapproj images
      if (!stereo_settings().match_files_prefix.empty() ||
          !stereo_settings().clean_match_files_prefix.empty()) 
        vw_throw(ArgumentErr() << "Options: --match-files-prefix and "
                 << "--clean-match-files-prefix do not work with mapprojected images.\n");
      
    } // End if dem_provided

    if (stereo_settings().corr_kernel[0]%2 == 0 ||
        stereo_settings().corr_kernel[1]%2 == 0   ){
      vw_throw(ArgumentErr() << "The entries of corr-kernel must be odd numbers.\n");
    }

    if (stereo_settings().subpixel_kernel[0]%2 == 0 ||
        stereo_settings().subpixel_kernel[1]%2 == 0   ){
      vw_throw(ArgumentErr() << "The entries of subpixel-kernel must be odd numbers.\n");
    }

    // Check SGM-related settings.

    vw::stereo::CorrelationAlgorithm stereo_alg
      = asp::stereo_alg_to_num(stereo_settings().stereo_algorithm);

    bool using_tiles = (stereo_alg > vw::stereo::VW_CORRELATION_BM ||
                        stereo_settings().alignment_method == "local_epipolar");

    bool using_sgm = (stereo_alg > vw::stereo::VW_CORRELATION_BM && 
        stereo_alg < vw::stereo::VW_CORRELATION_OTHER);

    if (!using_sgm) {
      if (stereo_settings().cost_mode == 3)
        vw_throw(ArgumentErr() << "Cannot use the census transform without SGM!\n" );
      if (stereo_settings().cost_mode == 4)
        vw_throw(ArgumentErr() << "Cannot use the ternary census transform without SGM!\n" );
    }
    
    if (stereo_settings().cost_mode > 4)
      vw_throw(ArgumentErr() << "Unknown value " << stereo_settings().cost_mode
               << " for cost-mode.\n");

    if (stereo_settings().min_triangulation_angle <= 0 &&
        stereo_settings().min_triangulation_angle != -1) {
      // This means the user modified it. Then it must be positive.
      vw_throw(ArgumentErr() << "The min triangulation angle must be positive.\n");
    }
    if (stereo_settings().min_triangulation_angle == -1) {
      // This means that the user did not set it. Set it to 0.
      // Deep inside StereoModel.cc it will be overwritten with some
      // positive value.
      // This is a bit awkward but is done so for backward compatibility.
      stereo_settings().min_triangulation_angle = 0;
    }

    if (opt.session->do_bathymetry()) {
      if (stereo_settings().refraction_index <= 1.0) 
        vw_throw(ArgumentErr() << "The water index of refraction to be used in "
                 << "bathymetry correction must be bigger than 1.\n");

      if (stereo_settings().bathy_plane == "") 
        vw_throw(ArgumentErr() << "The value of --bathy-plane was unspecified.\n");

      // Sanity check reading the bathy plane
      std::vector<BathyPlaneSettings> bathy_plane_set;
      read_bathy_plane_set(stereo_settings().bathy_plane, bathy_plane_set);
      
      if (opt.session->name() != "dg" &&
          opt.session->name() != "rpc" &&
          opt.session->name() != "dgmaprpc" &&
          opt.session->name() != "rpcmaprpc" &&
          opt.session->name() != "nadirpinhole" &&
          opt.session->name() != "pinholemappinhole")
        vw_throw(ArgumentErr() << "Bathymetry correction only works with dg, rpc, and "
                 << "nadirpinhole sessions, and mapprojected images for these. Got: "
                 << opt.session->name() << ".\n");

      if (stereo_settings().alignment_method != "homography"     &&
          stereo_settings().alignment_method != "affineepipolar" &&
          stereo_settings().alignment_method != "local_epipolar" &&
          stereo_settings().alignment_method != "none") 
        vw_throw(ArgumentErr() << "Bathymetry correction only works with alignment methods "
                 << "homography, affineepipolar, local_epipolar, and none.\n");
      
    }

    // Need the percentage to be more than 50 as we look at the range [100 - pct, pct].
    if (stereo_settings().outlier_removal_params[0] <= 50.0)
      vw_throw(ArgumentErr() << "The --outlier-removal-params percentage must be more than 50.\n");
    if (stereo_settings().outlier_removal_params[1] <= 0.0)
      vw_throw(ArgumentErr() << "The --outlier-removal-params factor must be positive.\n");

    if (stereo_settings().save_lr_disp_diff) {
      
      if (stereo_settings().xcorr_threshold < 0.0) 
        vw_throw(ArgumentErr() << "Must have a non-negative value of --xcorr-threshold "
                 << "to be able to use --save-left-right-disparity-difference.\n");

      if (stereo_alg >= vw::stereo::VW_CORRELATION_OTHER) 
        vw_throw(ArgumentErr() << "Can use --save-left-right-disparity-difference "
                 << "only with stereo algorithms asp_bm, asp_sgm, asp_mgm, and asp_final_mgm.\n");
    }

    // If later we perform piecewise adjustments, the cameras loaded
    // so far must not be adjusted. And we also can't just perform
    // stereo on cropped images, as we need the full disparity.
    if (stereo_settings().image_lines_per_piecewise_adjustment > 0) {
      // This check must come first as it implies adjusted cameras
      if ((stereo_settings().left_image_crop_win  != BBox2i(0, 0, 0, 0)) &&
          (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0)))
        vw_throw(ArgumentErr() << "Since we perform piecewise adjustments we "
                 << "need the full disparities, so --left-image-crop-win and  "
                 << "--right-image-crop-win cannot be used.\n");
      
      if (stereo_settings().piecewise_adjustment_interp_type != 1 &&
          stereo_settings().piecewise_adjustment_interp_type != 2)
        vw_throw(ArgumentErr() << "Interpolation type for piecewise "
                 << "adjustment can be only 1 or 2.\n");
    }

    // Camera checks
    if (!stereo_settings().correlator_mode) {
      try {
        // Note. Cameras are loaded just once, and repeated invocation of camera_models()
        // will not reload them. Hence this check does not incur a performance hit
        // due to loading of the cameras. 
        boost::shared_ptr<camera::CameraModel> camera_model1, camera_model2;
        opt.session->camera_models(camera_model1, camera_model2);
        
        Vector3 cam1_ctr = camera_model1->camera_center(Vector2());
        Vector3 cam2_ctr = camera_model2->camera_center(Vector2());
        Vector3 cam1_vec = camera_model1->pixel_to_vector(Vector2());
        Vector3 cam2_vec = camera_model2->pixel_to_vector(Vector2());
        // Do the cameras appear to be in the same location?
        if (norm_2(cam1_ctr - cam2_ctr) < 1e-3)
          vw_out(WarningMessage)
            << "Your cameras appear to be in the same location!\n"
            << "\tYou should double check your given camera\n"
            << "\tmodels as most likely stereo won't be able\n"
            << "\tto triangulate or perform epipolar rectification.\n";
        
        // Developer friendly help
        VW_OUT(DebugMessage,"asp") << "Camera 1 location: " << cam1_ctr << "\n"
                                   << "   in estimated Lon Lat Rad: "
                                   << cartography::xyz_to_lon_lat_radius_estimate(cam1_ctr) << "\n";
        VW_OUT(DebugMessage,"asp") << "Camera 2 location: " << cam2_ctr << "\n"
                                   << "   in estimated Lon Lat Rad: "
                                   << cartography::xyz_to_lon_lat_radius_estimate(cam2_ctr) << "\n";
        VW_OUT(DebugMessage,"asp") << "Camera 1 Pointing Dir: " << cam1_vec << "\n"
                                   << "      dot against pos: " << dot_prod(cam1_vec, cam1_ctr)
                                   << "\n";
        VW_OUT(DebugMessage,"asp") << "Camera 2 Pointing Dir: " << cam2_vec << "\n"
                                   << "      dot against pos: " << dot_prod(cam2_vec, cam2_ctr)
                                   << "\n";
        vw_out() << "Distance between camera centers in meters: "
                 << norm_2(cam1_ctr - cam2_ctr) << ".\n";
        
        // Can cameras triangulate to point at something in front of them?
        stereo::StereoModel model(camera_model1.get(), camera_model2.get());
        double error;
        Vector3 point = model(Vector2(), Vector2(), error);
        if (point != Vector3() // triangulation succeeded
            && ((dot_prod(cam1_vec, point - cam1_ctr) < 0) ||
                (dot_prod(cam2_vec, point - cam2_ctr) < 0)   )
            ){
          vw_out(WarningMessage)
            << "Your cameras appear to not to be pointing at the same location! "
            << "A test vector triangulated backwards through "
            << "the camera models. You should double check "
            << "your input models as most likely stereo won't "
            << "be able to triangulate.\n";
        }
        
      } catch (const std::exception& e) {
        // Don't throw an error here. There are legitimate reasons as to
        // why the first checks may fail. For example, the top left pixel
        // might not be valid on a map projected image. But notify the
        // user anyway. Make an exception for the piecewise adjustment checks.
        vw_out(DebugMessage,"asp") << e.what() << std::endl;
      }
    }
    
  } // End user_safety_checks

  // See if user's request to skip image normalization can be
  // satisfied.  This option is a speedup switch which is only meant
  // to work with with mapprojected images. It is also not documented.
  bool skip_image_normalization(ASPGlobalOptions const& opt){

    if (!stereo_settings().skip_image_normalization) 
      return false;
    
    bool crop_left  = (stereo_settings().left_image_crop_win  != BBox2i(0, 0, 0, 0));
    bool crop_right = (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0));

    // Respect user's choice for skipping the normalization of the input
    // images, if feasible.
    bool is_good = (!crop_left && !crop_right                    &&
                    stereo_settings().alignment_method == "none" &&
                    stereo_settings().cost_mode == 2             &&
                    has_tif_or_ntf_extension(opt.in_file1)       &&
                    has_tif_or_ntf_extension(opt.in_file2));

    if (!is_good) 
      vw_throw(ArgumentErr()
               << "Cannot skip image normalization unless there is no alignment, "
               << "no use of --left-image-crop-win and --right-image-crop-win, "
               << "the option --cost-mode is set to 2, and the input images have "
               << ".tif or .ntf extension.");

    return is_good;
  } // End function skip_image_normalization

  // Convert, for example, 'asp_mgm' to '2'. For ASP algorithms we
  // use the numbers 0 (BM), 1 (SGM), 2 (MGM), 3 (Final MGM).  For
  // external algorithms will have to examine closer the algorithm
  // string. This function has a Python analog in parallel_stereo.
  vw::stereo::CorrelationAlgorithm stereo_alg_to_num(std::string alg) {

    // Make it lowercase first
    boost::to_lower(alg);

    // Sanity check
    if (alg == "") 
      vw_throw(ArgumentErr() << "No stereo algorithm was specified.\n");
    
    if (alg.rfind("0", 0) == 0 || alg.rfind("asp_bm", 0) == 0) 
      return vw::stereo::VW_CORRELATION_BM;
    
    if (alg.rfind("1", 0) == 0 || alg.rfind("asp_sgm", 0) == 0) 
      return vw::stereo::VW_CORRELATION_SGM;
    
    if (alg.rfind("2", 0) == 0 || alg.rfind("asp_mgm", 0) == 0) 
      return vw::stereo::VW_CORRELATION_MGM;
    
    if (alg.rfind("3", 0) == 0 || alg.rfind("asp_final_mgm", 0) == 0) 
      return vw::stereo::VW_CORRELATION_FINAL_MGM;

    // Sanity check. Any numerical values except 0, 1, 2, 3 are not accepted.
    int num = atof(alg.c_str());
    if (num < 0 || num > 3) 
      vw_throw(ArgumentErr() << "Unknown algorithm: " << alg << ".\n");
    
    // An external stereo algorithm
    return vw::stereo::VW_CORRELATION_OTHER;
  }
  
} // end namespace asp
