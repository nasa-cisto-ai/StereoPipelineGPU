#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file mapproject.cc
///
/// This program will project a camera image onto a DEM using the
/// camera model.
/// TODO: This creates an RGBA output for an RGB input.
/// It should create an RBG output for RGB input,
/// and same for RGBA.

#include <vw/Cartography/Map2CamTrans.h>
#include <vw/Cartography/PointImageManipulation.h>
#include <vw/Cartography/CameraBBox.h>
#include <vw/Camera/PinholeModel.h>
#include <vw/Image/Algorithms2.h>
#include <vw/Image/Filter.h>

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/Core/StereoSettings.h>

using namespace vw;
using namespace vw::cartography;
namespace po = boost::program_options;
namespace fs = boost::filesystem;

/// The pixel type used for the DEM data
typedef PixelMask<float> DemPixelT;

struct Options : vw::GdalWriteOptions {
  // Input
  std::string dem_file, image_file, camera_file, output_file, stereo_session,
    bundle_adjust_prefix;
  bool isQuery, noGeoHeaderInfo, nearest_neighbor, parseOptions, dg_use_csm;
  bool multithreaded_model; // This is set based on the session type.
  bool enable_correct_velocity_aberration, enable_correct_atmospheric_refraction;
  
  // Keep a copy of the model here to not have to pass it around separately
  boost::shared_ptr<vw::camera::CameraModel> camera_model;
  
  // Settings
  std::string target_srs_string, output_type, metadata;
  double nodata_value, tr, mpp, ppd, datum_offset;
  BBox2 target_projwin, target_pixelwin;
};

void handle_arguments( int argc, char *argv[], Options& opt ) {
  po::options_description general_options("");
  double NaN = std::numeric_limits<double>::quiet_NaN();
  general_options.add_options()
    ("nodata-value",     po::value(&opt.nodata_value)->default_value(-32768),
     "No-data value to use unless specified in the input image.")
    ("t_srs",            po::value(&opt.target_srs_string)->default_value(""),
     "Specify the projection (PROJ.4 string). If not provided, use the one from the DEM.")
    ("tr",              po::value(&opt.tr)->default_value(NaN),
     "Set the output file resolution (ground sample distance) in target "
     "georeferenced units per pixel. This may be in degrees or meters, "
     "depending on your projection. The center of each output pixel "
     "will be at integer multiples of this grid size (hence the output "
     "image will extend for an additional half a pixel at each edge).")
    ("mpp",              po::value(&opt.mpp)->default_value(NaN),
     "Set the output file resolution in meters per pixel.")
    ("ppd",              po::value(&opt.ppd)->default_value(NaN),
     "Set the output file resolution in pixels per degree.")
    ("datum-offset",     po::value(&opt.datum_offset)->default_value(0),
     "When projecting to a datum instead of a DEM, use this elevation in meters from the datum.")
    ("query-projection", po::bool_switch(&opt.isQuery)->default_value(false),
     "Display the computed projection information and estimated ground sample distance (pixel size on the ground), and quit. Used by the mapproject script.")
    ("session-type,t",      po::value(&opt.stereo_session),
     "Select the stereo session type to use for processing. Usually the program can select this automatically by the file extension, except for xml cameras. See the doc for options.")
    ("t_projwin",        po::value(&opt.target_projwin),
     "Limit the map-projected image to this region, with the corners given in georeferenced coordinates (xmin ymin xmax ymax). Max is exclusive.")
    ("t_pixelwin",       po::value(&opt.target_pixelwin),
     "Limit the map-projected image to this region, with the corners given in pixels (xmin ymin xmax ymax). Max is exclusive.")
    ("bundle-adjust-prefix", po::value(&opt.bundle_adjust_prefix),
     "Use the camera adjustment obtained by previously running bundle_adjust with this output prefix.")
    ("ot",  po::value(&opt.output_type)->default_value("Float32"), "Output data type, when the input is single channel. Supported types: Byte, UInt16, Int16, UInt32, Int32, Float32. If the output type is a kind of integer, values are rounded and then clamped to the limits of that type. This option will be ignored for multi-channel images, when the output type is set to be the same as the input type.")
    ("nearest-neighbor", po::bool_switch(&opt.nearest_neighbor)->default_value(false),
     "Use nearest neighbor interpolation.  Useful for classification images.")
    ("mo",  po::value(&opt.metadata)->default_value(""), "Write metadata to the output file. Provide as a string in quotes if more than one item, separated by a space, such as 'VAR1=VALUE1 VAR2=VALUE2'. Neither the variable names nor the values should contain spaces.")
    ("no-geoheader-info", po::bool_switch(&opt.noGeoHeaderInfo)->default_value(false),
     "Do not write metadata information in the geoheader. See the doc for more info.")
    ("enable-correct-velocity-aberration", po::bool_switch(&opt.enable_correct_velocity_aberration)->default_value(false)->implicit_value(true),
     "Turn on velocity aberration correction for Optical Bar and non-ISIS linescan cameras. This option impairs the convergence of bundle adjustment.")
    ("enable-correct-atmospheric-refraction", po::bool_switch(&opt.enable_correct_atmospheric_refraction)->default_value(false)->implicit_value(true),
     "Turn on atmospheric refraction correction for Optical Bar and non-ISIS linescan cameras. This option impairs the convergence of bundle adjustment.")
    ("dg-use-csm", po::bool_switch(&opt.dg_use_csm)->default_value(false)->implicit_value(true),
     "Use the CSM model with DigitalGlobe linescan cameras (-t dg). No corrections are done for velocity aberration or atmospheric refraction.")
    ("parse-options", po::bool_switch(&opt.parseOptions)->default_value(false),
     "Parse the options and print the results. Used by the mapproject script.")
    ;
  general_options.add(vw::GdalWriteOptionsDescription(opt));
  
  po::options_description positional("");
  positional.add_options()
    ("dem",          po::value(&opt.dem_file))
    ("camera-image", po::value(&opt.image_file))
    ("camera-model", po::value(&opt.camera_file))
    ("output-image" , po::value(&opt.output_file));

  po::positional_options_description positional_desc;
  positional_desc.add("dem",          1);
  positional_desc.add("camera-image", 1);
  positional_desc.add("camera-model", 1);
  positional_desc.add("output-image", 1);

  std::string usage("[options] <dem> <camera-image> <camera-model> <output-image>\nInstead of the DEM file, a datum can be provided, such as\nWGS84, NAD83, NAD27, D_MOON, D_MARS, and MOLA.");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                            positional, positional_desc, usage,
                            allow_unregistered, unregistered);
  
  if ( !vm.count("dem") || !vm.count("camera-image") || !vm.count("camera-model") )
    vw_throw(ArgumentErr() << "Not all of the input DEM, image, and camera were specified.\n"
             << usage << general_options);
  
  // If exactly three files were passed in, the last one must be the output file and the image file
  // must contain the camera model.
  if ( !vm.count("output-image") && vm.count("camera-model") ) {
    opt.output_file = opt.camera_file;
    opt.camera_file = "";
  }

  if (asp::has_cam_extension(opt.output_file))
    vw_throw(ArgumentErr() << "The output file is a camera. Check your inputs.\n");

  if (opt.parseOptions) {
    // For the benefit of mapproject
    vw_out() << "dem," << opt.dem_file << std::endl;
    vw_out() << "image," << opt.image_file << std::endl;
    vw_out() << "camera," << opt.camera_file << std::endl;
    vw_out() << "output_file," << opt.output_file << std::endl;
    exit(0);
  }
  
  // Need this to be able to load adjusted camera models. That will happen
  // in the stereo session.
  asp::stereo_settings().bundle_adjust_prefix = opt.bundle_adjust_prefix;

  asp::stereo_settings().enable_correct_velocity_aberration
    = opt.enable_correct_velocity_aberration;
  asp::stereo_settings().enable_correct_atmospheric_refraction
    = opt.enable_correct_atmospheric_refraction;
  
  asp::stereo_settings().dg_use_csm = opt.dg_use_csm;
  
  if (fs::path(opt.dem_file).extension() != "") {
    // A path to a real DEM file was provided, load it!
    GeoReference dem_georef;
    bool has_georef = vw::cartography::read_georeference(dem_georef, opt.dem_file);
    if (!has_georef)
      vw_throw( ArgumentErr() << "There is no georeference information in: "
                              << opt.dem_file << ".\n" );

    // Make sure the user did not actually pass in an RGB image
    boost::shared_ptr<DiskImageResource> dem_rsrc = vw::DiskImageResourcePtr(opt.dem_file);
    ImageFormat dem_fmt = dem_rsrc->format();
    const int num_input_channels = num_channels(dem_fmt.pixel_format);
    if (num_input_channels > 2)
      vw_throw( ArgumentErr() << "Too many channels in: " << opt.dem_file << ".\n" );

    // Store the datum from the DEM
    // TODO (oalexan1): Storing the datum name is fragile.
    asp::stereo_settings().datum = dem_georef.datum().name(); 
  }

  return;
}

// If the output type is some type of int, round and clamp to this
// type, both the pixels and the nodata-value. Else write as it is.
template <class ImageT>
void write_parallel_type(std::string              const& filename,
                         ImageT                   const& image,
                         GeoReference             const& georef,
                         bool has_nodata, double nodata_val,
                         Options                  const& opt,
                         TerminalProgressCallback const& tpc) {

  typedef typename ImageT::pixel_type InputType;

  if (opt.output_type == "Float32") 
    write_parallel_cond(filename, image, georef, has_nodata, nodata_val, opt, tpc);
  else if (opt.output_type == "Byte") 
    write_parallel_cond(filename,
			per_pixel_filter(image, RoundAndClamp<uint8, InputType>()),
			georef, has_nodata,
			vw::round_and_clamp<uint8>(nodata_val),
			opt, tpc);
  else if (opt.output_type == "UInt16") 
    write_parallel_cond(filename,
			per_pixel_filter(image, RoundAndClamp<uint16, InputType>()),
                        georef, has_nodata,
			vw::round_and_clamp<uint16>(nodata_val),
			opt, tpc);
  else if (opt.output_type == "Int16") 
    write_parallel_cond(filename,
			per_pixel_filter(image, RoundAndClamp<int16, InputType>()),
                        georef, has_nodata,
			vw::round_and_clamp<int16>(nodata_val),
			opt, tpc);
  else if (opt.output_type == "UInt32") 
    write_parallel_cond(filename,
			per_pixel_filter(image, RoundAndClamp<uint32, InputType>()),
                        georef, has_nodata,
			vw::round_and_clamp<uint32>(nodata_val),
			opt, tpc);
  else if (opt.output_type == "Int32") 
    write_parallel_cond(filename,
			per_pixel_filter(image, RoundAndClamp<int32, InputType>()),
                        georef, has_nodata,
			vw::round_and_clamp<int32>(nodata_val),
			opt, tpc);
  else
    vw_throw( NoImplErr() << "Unsupported output type: " << opt.output_type << ".\n" );
  
}

template <class ImageT>
void write_parallel_cond(std::string              const& filename,
                         ImageViewBase<ImageT>    const& image,
                         GeoReference             const& georef,
                         bool has_nodata, double nodata_val,
                         Options                  const& opt,
                         TerminalProgressCallback const& tpc) {

  // Write names of the bundle adjust prefix, input image file, camera file,    
  // dem, and session type. Those will be used in StereoSession to load the
  // mapprojected image.
  // There is no difference between pinhole and nadirpinhole when it comes
  // to how mapprojection happens, that becomes important only in stereo.
  std::string session_type = opt.stereo_session;
  if (session_type == "isismapisis")
    session_type = "isis";
  if (session_type == "rpcmaprpc")
    session_type = "rpc";
  if (session_type == "pinholemappinhole" || session_type == "nadirpinhole")
    session_type = "pinhole";
  
  // Save some keywords that we will check later when using the mapprojected file

  std::map<std::string, std::string> keywords;
  if (!opt.noGeoHeaderInfo) {
    std::string prefix = asp::stereo_settings().bundle_adjust_prefix;;
    if (prefix == "") prefix = "NONE"; // to save the field, need to make it non-empty
    keywords["BUNDLE_ADJUST_PREFIX" ] = prefix;
    keywords["CAMERA_MODEL_TYPE" ]    = session_type;
    keywords["INPUT_IMAGE_FILE" ]     = opt.image_file;
    keywords["CAMERA_FILE" ]          = opt.camera_file;

    // Save the camera adjustment. That is an important record
    // for how the image got mapprojected and is good to keep.
    Vector3 t(0, 0, 0);
    vw::Quaternion<double> q(1, 0, 0, 0);
    vw::camera::AdjustedCameraModel * adj_cam
      = dynamic_cast<vw::camera::AdjustedCameraModel*>(opt.camera_model.get());
    if (adj_cam != NULL) {
      q = adj_cam->rotation();
      t = adj_cam->translation();
    }
    
    std::ostringstream osq;
    osq.precision(17);
    osq << q.w() << "," << q.x() << "," << q.y() << "," << q.z();
    keywords["ADJUSTMENT_QUATERNION"] = osq.str();
    
    std::ostringstream ost;
    ost.precision(17);
    ost << t.x() << "," << t.y() << "," << t.z();
    keywords["ADJUSTMENT_TRANSLATION"] = ost.str();

    keywords["DEM_FILE"] = opt.dem_file;
    
    // Parse keywords from the --mo option.
    asp::parse_append_metadata(opt.metadata, keywords);
  }
  
  bool has_georef = true;

  // ISIS is not thread safe so we must switch out based on what the session is.
  vw_out() << "Writing: " << filename << "\n";
  if (opt.multithreaded_model) {
    vw::cartography::block_write_gdal_image(filename, image.impl(), has_georef, georef,
                                has_nodata, nodata_val, opt, tpc, keywords);
  } else {
    vw::cartography::write_gdal_image(filename, image.impl(), has_georef, georef,
                          has_nodata, nodata_val, opt, tpc, keywords);
  }

}

/// Compute which camera pixel observes a DEM pixel.
Vector2 demPixToCamPix(Vector2i const& dem_pixel,
                      boost::shared_ptr<camera::CameraModel> const& camera_model,
                      ImageViewRef<DemPixelT> const& dem,
                      GeoReference const &dem_georef)
{
  Vector2 lonlat = dem_georef.point_to_lonlat(dem_georef.pixel_to_point(dem_pixel));
  //vw_out() << "lonlat = " << lonlat << std::endl;
  DemPixelT height = dem(dem_pixel[0], dem_pixel[1]);
  Vector3 xyz = dem_georef.datum().geodetic_to_cartesian
                    (Vector3(lonlat[0], lonlat[1], height.child()));
  //vw_out() << "xyz = " << xyz << std::endl;
  // Throws if the projection fails ???
  Vector2i camera_pixel = camera_model->point_to_pixel(xyz);
  //vw_out() << "camera_pixel = " << camera_pixel << std::endl;
  return camera_pixel;
}

/// Expand the ground BBox to contain all the corners of the DEM if they intersect the camera.
/// - TODO: This method still does not guarantee all points will be included in the bbox.
/// - TODO: This should probably take pixel validity into account!
void expandBboxToContainCornerIntersections(vw::CamPtr camera_model,
                                            ImageViewRef<DemPixelT> const& dem,
                                            GeoReference const &dem_georef,
                                            Vector2i const& image_size,
                                            BBox2 & bbox_on_ground) {
  // Each of the corners of the DEM
  std::vector<Vector2> dem_pixel_list(4);
  dem_pixel_list[0] = Vector2(0,            0           );
  dem_pixel_list[1] = Vector2(dem.cols()-1, 0           );
  dem_pixel_list[2] = Vector2(dem.cols()-1, dem.rows()-1);
  dem_pixel_list[3] = Vector2(0,            dem.rows()-1);

  for (int i=0; i<4; ++i) {
    try{
      // Project the DEM corner into the input image
      Vector2 dem_pixel = dem_pixel_list[i];
      Vector2 cam_pixel = demPixToCamPix(dem_pixel, camera_model, dem, dem_georef);

      // Get the point on the ground
      Vector2 groundLoc = dem_georef.pixel_to_point(dem_pixel);

      // If there was in intersection...
      if ( (cam_pixel.x() >= 0)              && (cam_pixel.y() > 0) &&
           (cam_pixel.x() <  image_size.x()) && (cam_pixel.y() < image_size.y()) ) {
        //Vector2 lonlat    = dem_georef.point_to_lonlat(dem_georef.pixel_to_point(dem_pixel));
        // Expand the ground bbox to contain in
        bbox_on_ground.grow(groundLoc);
        //vw_out() << "Grow --> " << groundLoc  << std::endl;
      }
      else{
        //vw_out() << "Miss! "  << std::endl;
      }
    }catch(...){
      //vw_out() << "Bad projection! "  << std::endl;
    } // If a point failed to project
  } // End loop through DEM points

}

/// Compute output georeference to use
void calc_target_geom(// Inputs
                      bool calc_target_res,
                      Vector2i const& image_size,
                      boost::shared_ptr<camera::CameraModel> const& camera_model,
                      ImageViewRef<DemPixelT> const& dem,
                      GeoReference const& dem_georef, 
                      bool datum_dem,
                      Options const & opt,
                      // Outputs
                      BBox2 & cam_box, GeoReference & target_georef){

  // Find the camera bbox and the target resolution unless user-supplied.
  // - This call returns the bounding box of the camera view on the ground.
  // - The bounding box is in units defined by dem_georef and might not be meters.
  // - auto_res is an estimate of the ground resolution visible by the camera.
  //   This is in a unit defined by dem_georef and also might not be meters.
  // - This call WILL intersect pixels outside the dem valid area!
  // - TODO: Modify this function to optionally disable intersection outside the DEM
  float auto_res = -1.0;  // will be updated
  bool quick = datum_dem; // The non-quick option does not make sense with huge DEMs.
  try {
    cam_box = camera_bbox(dem, dem_georef,
                          target_georef, 
                          camera_model,
                          image_size.x(), image_size.y(), auto_res, quick);
  } catch (std::exception const& e) {
    if (opt.target_projwin == BBox2() || calc_target_res) {
      vw_throw( ArgumentErr()
                << e.what() << "\n"
                << "Check your inputs. Or try specifying --t_projwin and --tr values.\n");
    }
  }

  // Use auto-calculated ground resolution if that option was selected
  double current_resolution;
  if (calc_target_res) {
    current_resolution = auto_res;
  } else {
    // Set the resolution from input options
    if (target_georef.is_projected()) {
      current_resolution = opt.mpp; // Use units of meters
    } else { // Not projected, GDC coordinates only.
      current_resolution = 1/opt.ppd; // Use units of degrees
                                      // Lat/lon degrees are different so we never want to do this!
    }
  }
  vw_out() << "Output pixel size: " << current_resolution << std::endl;

  // If an image bounding box (projected coordinates) was passed in,
  // override the camera's view on the ground with the custom box.
  // The user needs to know the georeference projected coordinate
  // system (using the query command) to do this.
  if ( opt.target_projwin != BBox2() ) {
    cam_box = opt.target_projwin;
    if ( cam_box.min().y() > cam_box.max().y() )
      std::swap( cam_box.min().y(), cam_box.max().y() );
    // The adjustments below are possibly to make the maximum
    // non-exclusive.
    cam_box.max().x() -= current_resolution; 
    cam_box.min().y() += current_resolution;
  }

  // In principle the corners of the projection box can be
  // arbitrary.  However, we will force them to be at integer
  // multiples of pixel dimensions. This is needed if we want to do
  // tiling, that is break the DEM into tiles, project on individual
  // tiles, and then combine the tiles nicely without seams into a
  // single projected image. The tiling solution provides a nice
  // speedup when dealing with ISIS images, when projection runs
  // only with one thread.
  double s = current_resolution;
  int min_x         = (int)round(cam_box.min().x() / s);
  int min_y         = (int)round(cam_box.min().y() / s);
  int output_width  = (int)round(cam_box.width()   / s);
  int output_height = (int)round(cam_box.height()  / s);
  cam_box = s * BBox2(min_x, min_y, output_width, output_height);

  // This transform is from pixel to projected coordinates
  Matrix3x3 T = target_georef.transform();
  // This polarity checking is to make sure the output has been
  // transposed after going through reprojection. Normally this is
  // the case. Yet with grid data from GMT, it is not.
  if ( T(0,0) < 0 ) // If X coefficient of affine transform is negative (cols go opposite direction from projected x coords)
    T(0,2) = cam_box.max().x();  // The maximum projected X coordinate is the starting offset
  else
    T(0,2) = cam_box.min().x(); // The minimum projected X coordinate is the starting offset
  T(0,0) =  current_resolution;  // Set col/X conversion to meters per pixel
  T(1,1) = -current_resolution;  // Set row/Y conversion to meters per pixel with a vertical flip (increasing row = down in Y)
  T(1,2) = cam_box.max().y();       // The maximum projected Y coordinate is the starting offset
  if ( target_georef.pixel_interpretation() ==
       GeoReference::PixelAsArea ) { // Meaning point [0][0] equals location (0.5, 0.5)
    T(0,2) -= 0.5 * current_resolution; // Apply a small shift to the offsets
    T(1,2) += 0.5 * current_resolution;
  }
  target_georef.set_transform(T); // Overwrite the existing transform in target_georef

  // Compute output image size in pixels using bounding box in output projected space
  BBox2i target_image_size = target_georef.point_to_pixel_bbox( cam_box );

  // Last adjustment, to ensure 0 0 is always in the box corner
  target_georef = crop(target_georef, target_image_size.min().x(), target_image_size.min().y());

  return;
}


/// Map project the image with a nodata value.  Used for single channel images.
template <class ImagePixelT, class Map2CamTransT>
void project_image_nodata(Options & opt,
                          GeoReference const& croppedGeoRef,
                          Vector2i     const& virtual_image_size,
                          BBox2i       const& croppedImageBB,
                          Map2CamTransT const& transform) {

    typedef PixelMask<ImagePixelT> ImageMaskPixelT;

    // Create handle to input image to be projected on to the map
    boost::shared_ptr<DiskImageResource> img_rsrc = 
          vw::DiskImageResourcePtr(opt.image_file);   

    // Update the nodata value from the input file if it is present.
    if (img_rsrc->has_nodata_read()) 
      opt.nodata_value = img_rsrc->nodata_read();

    bool            has_img_nodata = true;
    ImageMaskPixelT nodata_mask    = ImageMaskPixelT(); // invalid value for a PixelMask

    // TODO: This is a lot of code duplication, is there a better way?
    if (opt.nearest_neighbor) {
      write_parallel_type
        ( // Write to the output file
        opt.output_file,
        crop( // Apply crop (only happens if --t_pixelwin was specified)
              apply_mask
              ( // Handle nodata
              transform_nodata( // Apply the output from Map2CamTrans
                                create_mask(DiskImageView<ImagePixelT>(img_rsrc),
                                            opt.nodata_value), // Handle nodata
                                transform,
                                virtual_image_size[0],
                                virtual_image_size[1],
                                ValueEdgeExtension<ImageMaskPixelT>(nodata_mask),
                                NearestPixelInterpolation(), nodata_mask
                                ),
              opt.nodata_value
              ),
              croppedImageBB
              ),
        croppedGeoRef, has_img_nodata, opt.nodata_value, opt,
        TerminalProgressCallback("","")
        );
    } else {
      write_parallel_type
        ( // Write to the output file
        opt.output_file,
        crop( // Apply crop (only happens if --t_pixelwin was specified)
              apply_mask
              ( // Handle nodata
              transform_nodata( // Apply the output from Map2CamTrans
                                create_mask(DiskImageView<ImagePixelT>(img_rsrc),
                                            opt.nodata_value), // Handle nodata
                                transform,
                                virtual_image_size[0],
                                virtual_image_size[1],
                                ValueEdgeExtension<ImageMaskPixelT>(nodata_mask),
                                BicubicInterpolation(), nodata_mask
                                ),
              opt.nodata_value
              ),
              croppedImageBB
              ),
        croppedGeoRef, has_img_nodata, opt.nodata_value, opt,
        TerminalProgressCallback("","")
        );
    }

}

/// Map project the image with an alpha channel.  Used for multi-channel images.
template <class ImagePixelT, class Map2CamTransT>
void project_image_alpha(Options & opt,
                         GeoReference const& croppedGeoRef,
                         Vector2i     const& virtual_image_size,
                         BBox2i       const& croppedImageBB,
                         boost::shared_ptr<camera::CameraModel> const& camera_model,
                         Map2CamTransT const& transform) {
  
    // Create handle to input image to be projected on to the map
    boost::shared_ptr<DiskImageResource> img_rsrc = 
          vw::DiskImageResourcePtr(opt.image_file);   

    const bool        has_img_nodata    = false;
    const ImagePixelT transparent_pixel = ImagePixelT();

    // TODO: Is it possible to reduce code duplication?
    if (opt.nearest_neighbor) {
      write_parallel_type
        ( // Write to the output file
        opt.output_file,
        crop( // Apply crop (only happens if --t_pixelwin was specified)
              // Transparent pixels are inserted for nodata
              transform_nodata( // Apply the output from Map2CamTrans
                                DiskImageView<ImagePixelT>(img_rsrc),
                                transform,
                                virtual_image_size[0],
                                virtual_image_size[1],
                                ConstantEdgeExtension(),
                                NearestPixelInterpolation(), transparent_pixel),
              croppedImageBB),
        croppedGeoRef, has_img_nodata, opt.nodata_value, opt,
        TerminalProgressCallback("","")
        );
    } else {
      write_parallel_type
        ( // Write to the output file
        opt.output_file,
        crop( // Apply crop (only happens if --t_pixelwin was specified)
              // Transparent pixels are inserted for nodata
              transform_nodata( // Apply the output from Map2CamTrans
                                DiskImageView<ImagePixelT>(img_rsrc),
                                transform,
                                virtual_image_size[0],
                                virtual_image_size[1],
                                ConstantEdgeExtension(),
                                BicubicInterpolation(), transparent_pixel),
              croppedImageBB),
        croppedGeoRef, has_img_nodata, opt.nodata_value, opt,
        TerminalProgressCallback("",""));
    }

}

// The two "pick" functions below select between the Map2CamTrans and Datum2CamTrans
// transform classes which will be passed to the image projection function.
// - TODO: Is there a good reason for the transform classes to be CRTP instead of virtual?

template <class ImagePixelT>
void project_image_nodata_pick_transform(Options & opt,
                          GeoReference const& dem_georef,
                          GeoReference const& target_georef,
                          GeoReference const& croppedGeoRef,
                          Vector2i     const& image_size,
                          Vector2i     const& virtual_image_size,
                          BBox2i       const& croppedImageBB,
                          boost::shared_ptr<camera::CameraModel> const& camera_model) {
  const bool        call_from_mapproject = true;
  if (fs::path(opt.dem_file).extension() != "") {
    // A DEM file was provided
    return project_image_nodata<ImagePixelT>(opt, croppedGeoRef,
                                             virtual_image_size, croppedImageBB,
                                             Map2CamTrans(// Converts coordinates in DEM
                                                          // georeference to camera pixels
                                                          camera_model.get(), target_georef,
                                                          dem_georef, opt.dem_file, image_size,
                                                          call_from_mapproject,
                                                          opt.nearest_neighbor));
  } else {
    // A constant datum elevation was provided
    return project_image_nodata<ImagePixelT>(opt, croppedGeoRef,
                                             virtual_image_size, croppedImageBB,
                                             Datum2CamTrans
                                             (// Converts coordinates in DEM
                                              // georeference to camera pixels
                                              camera_model.get(), target_georef,
                                              dem_georef, opt.datum_offset, image_size,
                                              call_from_mapproject,
                                              opt.nearest_neighbor));
  }
}

template <class ImagePixelT>
void project_image_alpha_pick_transform(Options & opt,
                                        GeoReference const& dem_georef,
                                        GeoReference const& target_georef,
                                        GeoReference const& croppedGeoRef,
                                        Vector2i     const& image_size,
                                        Vector2i     const& virtual_image_size,
                                        BBox2i       const& croppedImageBB,
                                        boost::shared_ptr<camera::CameraModel> const&
                                        camera_model) {
  
  const bool        call_from_mapproject = true;
  if (fs::path(opt.dem_file).extension() != "") {
    // A DEM file was provided
    return project_image_alpha<ImagePixelT>(opt, croppedGeoRef,
                                            virtual_image_size, croppedImageBB, camera_model, 
                                            Map2CamTrans(// Converts coordinates in DEM
                                                         // georeference to camera pixels
                                                         camera_model.get(), target_georef,
                                                         dem_georef, opt.dem_file, image_size,
                                                         call_from_mapproject,
                                                         opt.nearest_neighbor));
  } else {
    // A constant datum elevation was provided
    return project_image_alpha<ImagePixelT>(opt, croppedGeoRef,
                                            virtual_image_size, croppedImageBB, camera_model, 
                                            Datum2CamTrans(// Converts coordinates in DEM
                                                           // georeference to camera pixels
                                                           camera_model.get(), target_georef,
                                                           dem_georef, opt.datum_offset, image_size,
                                                           call_from_mapproject,
                                                           opt.nearest_neighbor));
  }
}

int main(int argc, char* argv[]) {

  Options opt;
  try {
    handle_arguments(argc, argv, opt);
  
    // TODO: Replace this using the new CameraModelLoader functions. But those
    // may not have the session guessing logic.

    // We create a stereo session where both of the cameras and images
    // are the same, because we want to take advantage of the stereo
    // pipeline's ability to generate camera models for various
    // missions.  Hence, we create two identical camera models, but only one is used.
    typedef boost::scoped_ptr<asp::StereoSession> SessionPtr;
    SessionPtr session(asp::StereoSessionFactory::create
                       (opt.stereo_session, // in-out
                        opt,
                        opt.image_file, opt.image_file, // The same file is passed in twice
                        opt.camera_file, opt.camera_file,
                        opt.output_file,
                        "", // Do not use a DEM to not make the session mapprojected
                        false)); // Do not allow promotion from normal to map projected session
    
    if ( opt.output_file.empty() )
      vw_throw( ArgumentErr() << "Missing output filename.\n" );

    // Additional checks once the stereo session is determined.
    
    if (opt.stereo_session == "dg" || opt.stereo_session == "perusat")
      vw_out(WarningMessage) << "Images map-projected using the '" << opt.stereo_session
                             << "' camera model cannot be used later for stereo. "
                             << "If that is desired, please run mapproject with "
                             << "'-t rpc' and a camera file having an RPC model.\n";

    // If nothing else works
    // TODO(oalexan1): Likely StereoSessionFactory already have this logic.
    if (boost::iends_with(boost::to_lower_copy(opt.camera_file), ".xml") &&
         opt.stereo_session == "" )
      opt.stereo_session = "rpc";

    // Initialize the camera model
    opt.camera_model = session->camera_model(opt.image_file, opt.camera_file);

    opt.multithreaded_model = session->supports_multi_threading();
      
    {
      // Safety check that the users are not trying to map project map
      // projected images. This should not be an error as sometimes
      // even raw images have some half-baked georeference attached to them.
      GeoReference dummy_georef;
      bool has_georef = vw::cartography::read_georeference( dummy_georef, opt.image_file );
      if (has_georef)
        vw_out(WarningMessage) << "Your input camera image is already map-"
                               << "projected. The expected input is required "
                               << "to be unprojected or raw camera imagery.\n";
    }

    // Load the DEM
    bool datum_dem = false;
    GeoReference dem_georef;
    ImageViewRef<DemPixelT> dem;
    if (fs::path(opt.dem_file).extension() != "") {
      // A path to a real DEM file was provided, load it!

      bool has_georef = vw::cartography::read_georeference(dem_georef, opt.dem_file);
      if (!has_georef)
        vw_throw( ArgumentErr() << "There is no georeference information in: "
                  << opt.dem_file << ".\n" );

      boost::shared_ptr<DiskImageResource> dem_rsrc(DiskImageResourcePtr(opt.dem_file));

      // If we have a nodata value, create a mask.
      DiskImageView<float> dem_disk_image(opt.dem_file);
      if (dem_rsrc->has_nodata_read()){
        dem = create_mask(dem_disk_image, dem_rsrc->nodata_read());
      }else{
        dem = pixel_cast<DemPixelT>(dem_disk_image);
      }      
    } else {
      // Projecting to a datum instead of a DEM
      datum_dem = true;
      std::string datum_name = opt.dem_file;

      // Use the camera center to determine whether to center the fake DEM on 0 or 180.
      Vector3 cam_ctr = opt.camera_model->camera_center(Vector2());
      Vector3 llr_camera_loc = cartography::XYZtoLonLatRadEstimateFunctor::apply(cam_ctr);
      float lonstart = 0;
      if ((llr_camera_loc[0] < 0) && (llr_camera_loc[0] > -180))
        lonstart = -180;
      dem_georef = GeoReference(Datum(datum_name),
                                // Need adjustments to work at boundaries!
                                vw::Matrix3x3(1,  0, lonstart-0.5,
                                              0, -1, 90+0.5,
                                              0,  0,  1) );
      dem = constant_view(PixelMask<float>(opt.datum_offset), 360.0, 180.0);
      vw_out() << "\t--> Using flat datum \"" << datum_name << "\" as elevation model.\n";
      //std::cout << "dem_georef = " << dem_georef << std::endl;
    }
    // Finished setting up the datum

    // Read projection. Work out output bounding box in points using original camera model.
    GeoReference target_georef = dem_georef;

    // User specified the proj4 string for the output georeference
    if (opt.target_srs_string != ""){
      bool  have_user_datum = false, have_input_georef = false;
      Datum user_datum;
      asp::set_srs_string(opt.target_srs_string, have_user_datum, user_datum,
                          have_input_georef, target_georef);
    }

    // Find the target resolution based --tr, --mpp, and --ppd if provided. Do
    // the math to convert pixel-per-degree to meter-per-pixel and vice-versa.
    int sum = (!std::isnan(opt.tr)) + (!std::isnan(opt.mpp)) + (!std::isnan(opt.ppd));
    if (sum >= 2)
      vw_throw( ArgumentErr() << "Must specify at most one of the options: --tr, --mpp, --ppd.\n" );

    double radius = target_georef.datum().semi_major_axis();
    if ( !std::isnan(opt.tr) ){ // --tr was set
      if (target_georef.is_projected()) {
        if (std::isnan(opt.mpp)) opt.mpp = opt.tr; // User must have provided be meters per pixel
      }else {
        if (std::isnan(opt.ppd)) opt.ppd = 1.0/opt.tr; // User must have provided degrees per pixel
      }
    }
    
    if (!std::isnan(opt.mpp)){ // Meters per pixel was set
      if (std::isnan(opt.ppd)) opt.ppd = 2.0*M_PI*radius/(360.0*opt.mpp);
    }
    if (!std::isnan(opt.ppd)){ // Pixels per degree was set
      if (std::isnan(opt.mpp)) opt.mpp = 2.0*M_PI*radius/(360.0*opt.ppd);
    }
    
    bool user_provided_resolution = (!std::isnan(opt.ppd));
    bool     calc_target_res = !user_provided_resolution;
    Vector2i image_size      = vw::file_image_size(opt.image_file);
    BBox2    cam_box;
    calc_target_geom(// Inputs
                     calc_target_res, image_size, opt.camera_model,
                     dem, dem_georef, datum_dem,
                     // Outputs
                     opt, cam_box, target_georef);

    // Set a high precision, as the numbers can come out big for UTM
    vw_out() << std::setprecision(17) << "Projected space bounding box: " << cam_box << std::endl;

    // Compute output image size in pixels using bounding box in output projected space
    BBox2i target_image_size = target_georef.point_to_pixel_bbox( cam_box );

    vw_out() << std::setprecision(17) << "Image box: " << target_image_size << std::endl;
    
    // Very important note: this box may be in the middle of the
    // image.  However, the virtual image we create with
    // transform_nodata() below is assumed to start at (0, 0), and in
    // target_georef we assume the same thing. Hence, its width and
    // height are going to be the max values of target_image_size.
    // There is no performance hit here, since that potentially huge
    // image is never actually realized, we crop it as seen below
    // before finding its pixels. This could be made less confusing.
    int virtual_image_width  = target_image_size.max().x();
    int virtual_image_height = target_image_size.max().y();

    // Shrink output image BB if an output image BB was passed in
    GeoReference croppedGeoRef  = target_georef;
    BBox2i       croppedImageBB = target_image_size;
    if ( opt.target_pixelwin != BBox2() ) {
      // Replace with passed in bounding box
      croppedImageBB = opt.target_pixelwin;

      // Update output georeference to match the reduced image size
      croppedGeoRef = vw::cartography::crop(target_georef, croppedImageBB);
    }
    //vw_out() << "croppedImageBB = " << croppedImageBB << std::endl;
    //vw_out() << "\nCROPPED georeference:\n"        << croppedGeoRef << std::endl;

    // Important: Don't modify the line below, we count on it in mapproject.in.
    vw_out() << "Output image size:\n";
    vw_out() << std::setprecision(17) << "(width: " << virtual_image_width
             << " height: " << virtual_image_height << ")" << std::endl;

    if (opt.isQuery){ // Quit before we do any image work
      vw_out() << "Query finished, exiting mapproject tool.\n";
      return 0;
    }

    // For certain pinhole camera models the reverse check can make map projection very slow,
    // so we disable it here.  The check is very important for computing the bounding box safely
    // but we don't really need it when projecting the pixels back in to the camera.
    boost::shared_ptr<vw::camera::PinholeModel> pinhole_ptr = 
                boost::dynamic_pointer_cast<vw::camera::PinholeModel>(opt.camera_model);
    if (pinhole_ptr)
      pinhole_ptr->set_do_point_to_pixel_check(false);

    // Determine the pixel type of the input image
    boost::shared_ptr<DiskImageResource> image_rsrc = vw::DiskImageResourcePtr(opt.image_file);
    ImageFormat image_fmt = image_rsrc->format();
    const int num_input_channels = num_channels(image_fmt.pixel_format);

    // Prepare output directory
    vw::create_out_dir(opt.output_file);

    // Redirect to the correctly typed function to perform the actual map projection.
    // - Must correspond to the type of the input image.
    if (image_fmt.pixel_format == VW_PIXEL_RGB) {

      // We can't just use float for everything or the output will be cast
      //  into the -1 to 1 range which is probably not desired.
      // - Always use an alpha channel with RGB images.
      switch(image_fmt.channel_type) {
      case VW_CHANNEL_UINT8:
        project_image_alpha_pick_transform<PixelRGBA<uint8>>(opt, dem_georef, target_georef,
                                                             croppedGeoRef, image_size, 
                                                             Vector2i(virtual_image_width,
                                                                      virtual_image_height),
                                                             croppedImageBB, opt.camera_model);
        break;
      case VW_CHANNEL_INT16:
        project_image_alpha_pick_transform<PixelRGBA<int16>>(opt, dem_georef, target_georef,
                                                             croppedGeoRef, image_size, 
                                                             Vector2i(virtual_image_width,
                                                                      virtual_image_height),
                                                             croppedImageBB, opt.camera_model);
        break;
      case VW_CHANNEL_UINT16:
        project_image_alpha_pick_transform<PixelRGBA<uint16>>(opt, dem_georef, target_georef,
                                                              croppedGeoRef, image_size, 
                                                              Vector2i(virtual_image_width,
                                                                       virtual_image_height),
                                                              croppedImageBB, opt.camera_model);
        break;
      default:
        project_image_alpha_pick_transform<PixelRGBA<float32>>(opt, dem_georef, target_georef,
                                                               croppedGeoRef, image_size, 
                                                               Vector2i(virtual_image_width,
                                                                        virtual_image_height),
                                                               croppedImageBB, opt.camera_model);
        break;
      };
      
    } else {
      // If the input image is not RGB, only single channel images are supported.
      if (num_input_channels != 1 || image_fmt.planes != 1)
        //vw_throw( ArgumentErr() << "Input images must be single channel or RGB!\n" );
        vw_out() << "Detected multi-band image. Only the first band will be used. The pixels will be interpreted as float.\n";
      // This will cast to float but will not rescale the pixel values.
      project_image_nodata_pick_transform<float>(opt, dem_georef, target_georef, croppedGeoRef,
                                                 image_size, 
                           Vector2i(virtual_image_width, virtual_image_height),
                           croppedImageBB, opt.camera_model);
    } 
    // Done map projecting!

  } ASP_STANDARD_CATCHES;

  return 0;
}




