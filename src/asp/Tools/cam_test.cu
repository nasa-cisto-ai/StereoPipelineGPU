#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Tool to compare two camera models for the same image. For example,
// compare ISIS to CSM, linescan to RPC (for DG, PeruSat, or
// Pleiades), Optical bar vs pinhole (with the latter created with
// convert_pinhole_model).

// For each camera model find the camera center and ray direction at a
// set of sampled pixels, then by projecting pixels to the ground
// using the cam1 camera and back-projecting the resulting points into
// the cam2 camera, then doing this in reverse.

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Sessions/StereoSession.h>
#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/Camera/RPCModel.h>
#include <asp/Camera/CsmModel.h>
#include <asp/IsisIO/IsisCameraModel.h>
#include <asp/Camera/Covariance.h>

#include <vw/Core/Stopwatch.h>

using namespace vw;
using namespace vw::cartography;
namespace po = boost::program_options;
namespace fs = boost::filesystem;

typedef boost::scoped_ptr<asp::StereoSession> SessionPtr;

struct Options : vw::GdalWriteOptions {
  std::string image_file, cam1_file, cam2_file, session1, session2, bundle_adjust_prefix, datum;
  int sample_rate; // use one out of these many pixels
  double subpixel_offset, height_above_datum;
  bool enable_correct_velocity_aberration, enable_correct_atmospheric_refraction,
    print_per_pixel_results, dg_use_csm, dg_vs_csm, test_error_propagation;
  vw::Vector2 single_pixel;

  Options() {}
};

void handle_arguments(int argc, char *argv[], Options& opt) {

  double nan = std::numeric_limits<double>::quiet_NaN();
  po::options_description general_options("General options");
  general_options.add_options()
    ("image", po::value(&opt.image_file),  "Image file.")
    ("cam1",  po::value(&opt.cam1_file),   "Camera 1 file.")
    ("cam2",  po::value(&opt.cam2_file),   "Camera 2 file.")
    ("session1", po::value(&opt.session1),
     "Session to use for camera 1 (if not provided it will be guessed).")
    ("session2", po::value(&opt.session2),
     "Session to use for camera 2 (if not provided it will be guessed).")
    ("sample-rate",   po::value(&opt.sample_rate)->default_value(100),
     "Use one out of these many pixels when sampling the image.")
    ("subpixel-offset",   po::value(&opt.subpixel_offset)->default_value(0.0),
     "Add to each integer pixel this offset (in x and y) when sampling the image.")
    ("single-pixel",   po::value(&opt.single_pixel)->default_value(Vector2(nan, nan)),
     "Instead of sampling pixels from the image use only this pixel.")
    ("print-per-pixel-results", po::bool_switch(&opt.print_per_pixel_results)->default_value(false)->implicit_value(true),
     "Print the results at each pixel.")
    ("height-above-datum",   po::value(&opt.height_above_datum)->default_value(0.0),
     "Let the ground be obtained from the datum for this camera by "
     "adding to its radii this value (the units are meters).")
    ("datum", po::value(&opt.datum),
     "Set the datum. This will override the datum from the input cameras. Usually needed "
     "only for Pinhole cameras for non-Earth planets, when the camera does not have "
     "the datum information. Options: WGS_1984, D_MOON (1,737,400 meters), "
     "D_MARS (3,396,190 meters), MOLA (3,396,000 meters), NAD83, WGS72, "
     "and NAD27. Also accepted: Earth (=WGS_1984), Mars (=D_MARS), Moon (=D_MOON).")
    ("enable-correct-velocity-aberration", po::bool_switch(&opt.enable_correct_velocity_aberration)->default_value(false)->implicit_value(true),
     "Turn on velocity aberration correction for Optical Bar and non-ISIS linescan cameras. This option impairs the convergence of bundle adjustment.")
    ("enable-correct-atmospheric-refraction", po::bool_switch(&opt.enable_correct_atmospheric_refraction)->default_value(false)->implicit_value(true),
     "Turn on atmospheric refraction correction for Optical Bar and non-ISIS linescan cameras. This option impairs the convergence of bundle adjustment.")
    ("dg-use-csm", po::bool_switch(&opt.dg_use_csm)->default_value(false)->implicit_value(true),
     "Use the CSM model with DigitalGlobe linescan cameras (-t dg). No corrections are done for velocity aberration or atmospheric refraction.")
    ("dg-vs-csm", po::bool_switch(&opt.dg_vs_csm)->default_value(false)->implicit_value(true),
     "Compare projecting into the camera without and with using the CSM model for Digital Globe.")
    ("bundle-adjust-prefix", po::value(&opt.bundle_adjust_prefix),
     "Adjust the cameras using this prefix.")
    ("test-error-propagation", po::bool_switch(&opt.test_error_propagation)->default_value(false)->implicit_value(true),
     "Test computing the stddev (see --propagate-errors). This is an undocumented developer option.")
    ;
  general_options.add(vw::GdalWriteOptionsDescription(opt));

  po::options_description positional("");
  po::positional_options_description positional_desc;

  std::string usage("--image <image file> --cam1 <camera 1 file> --cam2 <camera 2 file> "
                    "[other options]");

  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                            positional, positional_desc, usage,
                            allow_unregistered, unregistered);

  if (opt.image_file == "" || opt.cam1_file == "" || opt.cam2_file == "")
    vw_throw(ArgumentErr() << "Not all inputs were specified.\n" << usage << general_options);

  if (opt.sample_rate <= 0)
    vw_throw(ArgumentErr() << "The sample rate must be positive.\n" << usage << general_options);

  asp::stereo_settings().enable_correct_velocity_aberration
    = opt.enable_correct_velocity_aberration;
  asp::stereo_settings().enable_correct_atmospheric_refraction
    = opt.enable_correct_atmospheric_refraction;
  asp::stereo_settings().dg_use_csm = opt.dg_use_csm;

  // Need this to be able to load adjusted camera models. This must be set
  // before loading the cameras.
  asp::stereo_settings().bundle_adjust_prefix = opt.bundle_adjust_prefix;

  if (opt.test_error_propagation) {
    if (!asp::stereo_settings().dg_use_csm) {
      vw_out() << "Enabling option --dg-use-csm as point cloud stddev will be computed.\n";
      asp::stereo_settings().dg_use_csm = true;
    }
    asp::stereo_settings().propagate_errors = true;
  }
}

// Sort the diffs and print some stats
void print_diffs(std::string const& tag, std::vector<double> & diffs) {
  std::sort(diffs.begin(), diffs.end());

  vw_out() << "\n";

  if (diffs.empty()) {
    vw_out() << "Empty list of diffs for: " << tag << "\n";
    return;
  }

  vw_out() << tag << "\n";
  vw_out() << "Min:    " << diffs[0] << "\n";
  vw_out() << "Median: " << diffs[diffs.size()/2] << "\n";
  vw_out() << "Max:    " << diffs.back() << "\n";
}

void testErrorPropagation(Options const& opt,
                          vw::cartography::Datum const& datum,
                          vw::CamPtr cam1_model,
                          vw::CamPtr cam2_model) {

  double major_axis = datum.semi_major_axis() + opt.height_above_datum;
  double minor_axis = datum.semi_minor_axis() + opt.height_above_datum;

  // Try to find a pair of pixels corresponding to same tri point,
  // within image bounds
  vw::Vector2 pix1, pix2;
  Vector3 triPt;
  for (int i = 0; i < 20; i++) {

    pix1 = Vector2(i * 1000, i * 1000);
    Vector3 cam1_dir = cam1_model->pixel_to_vector(pix1);
    Vector3 cam1_ctr = cam1_model->camera_center(pix1);

    // Shoot a ray from the cam1 camera, intersect it with the
    // given height above datum
    triPt = vw::cartography::datum_intersection(major_axis, minor_axis,
                                                      cam1_ctr, cam1_dir);

    // Project to second camera
    pix2 = cam2_model->point_to_pixel(triPt);

    if (pix2.x() > 0 && pix2.y() > 0)
      break;
  }

  std::cout << "Left pixel:  " << pix1 << std::endl;
  std::cout << "Right pixel: " << pix2 << std::endl;

  vw::Vector2 ans = asp::propagateCovariance(triPt, datum,
                                             cam1_model.get(), cam2_model.get(),
                                             pix1, pix2);
  std::cout << "Horizontal and vertical stddev: " << ans << std::endl;
}

int main(int argc, char *argv[]) {

  Options opt;
  try {
    handle_arguments(argc, argv, opt);

    // Load cam1
    std::string out_prefix;
    std::string default_session1 = opt.session1; // save it before it changes
    SessionPtr cam1_session(asp::StereoSessionFactory::create
                               (opt.session1, // may change
                                opt,
                                opt.image_file, opt.image_file,
                                opt.cam1_file, opt.cam1_file,
                                out_prefix));
    boost::shared_ptr<vw::camera::CameraModel> cam1_model
      = cam1_session->camera_model(opt.image_file, opt.cam1_file);

    // Load cam2
    std::string default_session2 = opt.session2; // save it before it changes
    SessionPtr cam2_session(asp::StereoSessionFactory::create
                           (opt.session2, // may change
                            opt,
                            opt.image_file, opt.image_file,
                            opt.cam2_file, opt.cam2_file,
                            out_prefix));
    boost::shared_ptr<vw::camera::CameraModel> cam2_model
      = cam2_session->camera_model(opt.image_file, opt.cam2_file);

    vw::cartography::Datum datum;
    if (opt.datum == "") {
      // Auto-guess the datum, this is the default
      bool use_sphere_for_non_earth = true;
      datum = cam1_session->get_datum(cam1_model.get(), use_sphere_for_non_earth);

      // Sanity check
      vw::cartography::Datum datum2 = cam2_session->get_datum(cam2_model.get(), 
                                                              use_sphere_for_non_earth);
      if (datum.semi_major_axis() != datum2.semi_major_axis() ||
          datum.semi_minor_axis() != datum2.semi_minor_axis())
            vw::vw_out(vw::WarningMessage) << "The two cameras have different datums:\n" 
                                           << datum << "\n" << datum2 << "\n"
                                           << "Consider using the --datum option.\n";
    } else {
      // Use the datum specified by the user
      datum.set_well_known_datum(opt.datum);
    }
    vw_out() << "Using datum: " << datum << std::endl;

    // Sanity check
    if (norm_2(cam1_model->camera_center(Vector2())) < datum.semi_major_axis() ||
        norm_2(cam2_model->camera_center(Vector2())) < datum.semi_major_axis())   
            vw::vw_out(vw::WarningMessage) << "First or second camera center is below "
            << "the datum semi-major axis. Check your data. Consider using "
            << "the --datum and/or --height-above-datum options.\n"; 

    if (opt.session1 == opt.session2 && (default_session1 == "" || default_session2 == ""))
      vw_throw(ArgumentErr() << "The session names for both cameras "
               << "were guessed as: '" << opt.session1 << "'. It is suggested that they be "
               << "explicitly specified using --session1 and --session2.\n");

    if (opt.test_error_propagation && opt.session1 == "dg" && opt.session2 == "dg") {
      testErrorPropagation(opt, datum, cam1_model, cam2_model);
      return 0;
    }

    // Find the input image dimensions
    int image_cols = 0, image_rows = 0;
    try {
      DiskImageView<float> image(opt.image_file);
      image_cols = image.cols();
      image_rows = image.rows();
    } catch(const std::exception& e) {
      // For CSM-to-CSM ground-to-image and image-to-ground comparisons only,
      // the camera has the dimensions if the .cub image is missing.
      asp::CsmModel * csm_model
        = dynamic_cast<asp::CsmModel*>(vw::camera::unadjusted_model(cam1_model.get()));
      if (csm_model != NULL) {
        image_cols = csm_model->get_image_size()[0];
        image_rows = csm_model->get_image_size()[1];
      } else {
        vw::vw_throw(ArgumentErr() << e.what());
      }
    }

    vw_out() << "Image dimensions: " << image_cols << ' ' << image_rows << std::endl;

    bool single_pix = !std::isnan(opt.single_pixel[0]) && !std::isnan(opt.single_pixel[1]);

    Stopwatch sw;
    sw.start();

    double major_axis = datum.semi_major_axis() + opt.height_above_datum;
    double minor_axis = datum.semi_minor_axis() + opt.height_above_datum;
    // Iterate over the image
    std::vector<double> ctr_diff, dir_diff, cam1_to_cam2_diff, cam2_to_cam1_diff, dg_vs_csm_diff;
    for (int col = 0; col < image_cols; col += opt.sample_rate) {
      for (int row = 0; row < image_rows; row += opt.sample_rate) {

        Vector2 image_pix(col + opt.subpixel_offset, row + opt.subpixel_offset);

        if (single_pix)
          image_pix = opt.single_pixel;

        if (opt.print_per_pixel_results || single_pix)
          vw_out() << "Pixel: " << image_pix << "\n";

        Vector3 cam1_ctr = cam1_model->camera_center(image_pix);
        Vector3 cam2_ctr = cam2_model->camera_center(image_pix);
        ctr_diff.push_back(norm_2(cam1_ctr - cam2_ctr));

        if (opt.print_per_pixel_results)
          vw_out() << "Camera center diff: " << ctr_diff.back() << std::endl;

        Vector3 cam1_dir = cam1_model->pixel_to_vector(image_pix);
        Vector3 cam2_dir = cam2_model->pixel_to_vector(image_pix);
        dir_diff.push_back(norm_2(cam1_dir - cam2_dir));

        if (opt.print_per_pixel_results)
          vw_out() << "Camera direction diff: " << dir_diff.back() << std::endl;

        // Shoot a ray from the cam1 camera, intersect it with the
        // given height above datum, and project it back into the cam2
        // camera.
        Vector3 xyz = vw::cartography::datum_intersection(major_axis, minor_axis,
                                                          cam1_ctr, cam1_dir);

        Vector2 cam2_pix = cam2_model->point_to_pixel(xyz);
        cam1_to_cam2_diff.push_back(norm_2(image_pix - cam2_pix));

        if (opt.print_per_pixel_results)
          vw_out() << "cam1 to cam2 pixel diff: " << image_pix - cam2_pix << std::endl;

        if (opt.dg_vs_csm) {
          asp::stereo_settings().dg_use_csm = !asp::stereo_settings().dg_use_csm;
          Vector2 cam2_pix2 = cam2_model->point_to_pixel(xyz);
          asp::stereo_settings().dg_use_csm = !asp::stereo_settings().dg_use_csm;
          dg_vs_csm_diff.push_back(norm_2(cam2_pix - cam2_pix2));
        }

        // Shoot a ray from the cam2 camera, intersect it with the
        // given height above the datum, and project it back into the
        // cam1 camera.
        xyz = vw::cartography::datum_intersection(major_axis, minor_axis,
                                                  cam2_ctr, cam2_dir);
        Vector2 cam1_pix = cam1_model->point_to_pixel(xyz);
        cam2_to_cam1_diff.push_back(norm_2(image_pix - cam1_pix));

        if (opt.print_per_pixel_results)
          vw_out() << "cam2 to cam1 pixel diff: " << image_pix - cam1_pix << "\n\n";

        if (opt.dg_vs_csm) {
          asp::stereo_settings().dg_use_csm = !asp::stereo_settings().dg_use_csm;
          Vector2 cam1_pix2 = cam1_model->point_to_pixel(xyz);
          asp::stereo_settings().dg_use_csm = !asp::stereo_settings().dg_use_csm;
          dg_vs_csm_diff.push_back(norm_2(cam1_pix - cam1_pix2));
        }

        if (single_pix)
          break;
      }

      if (single_pix)
        break;
    }

    sw.stop();
    vw_out() << "Number of samples used: " << ctr_diff.size() << "\n";

    print_diffs("cam1 to cam2 camera direction diff norm", dir_diff);
    print_diffs("cam1 to cam2 camera center diff (meters)", ctr_diff);
    print_diffs("cam1 to cam2 pixel diff", cam1_to_cam2_diff);
    print_diffs("cam2 to cam1 pixel diff", cam2_to_cam1_diff);
    if (opt.dg_vs_csm)
    print_diffs("dg vs csm pixel diff", dg_vs_csm_diff);

    double elapsed_sec = sw.elapsed_seconds();
    vw_out() << "\nElapsed time per sample: " << 1e+6 * elapsed_sec/ctr_diff.size()
             << " milliseconds.\n";

    if (elapsed_sec < 5)
      vw_out() << "It is suggested to adjust the sample rate to produce more samples "
               << "if desired to evaluate more accurately the elapsed time per sample.\n";

  } ASP_STANDARD_CATCHES;

  return 0;
}
