#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Create a pinhole or optical bar camera model based on intrinsics, image corner
// coordinates, and, optionally, a DEM of the area.

#include <vw/FileIO/DiskImageView.h>
#include <vw/Core/StringUtils.h>
#include <vw/Camera/PinholeModel.h>
#include <vw/Camera/CameraUtilities.h>
#include <vw/Cartography/Datum.h>
#include <vw/Cartography/GeoReference.h>
#include <vw/Cartography/CameraBBox.h>
#include <vw/Math/LevenbergMarquardt.h>
#include <vw/Math/Geometry.h>
#include <vw/Stereo/StereoModel.h>
#include <vw/Camera/OpticalBarModel.h>
#include <asp/Core/Common.h>
#include <asp/Core/Macros.h>
#include <asp/Core/FileUtils.h>
#include <asp/Core/PointUtils.h>
#include <asp/Core/EigenUtils.h>
#include <asp/Camera/CameraResectioning.h>
#include <asp/Sessions/StereoSession.h>
#include <asp/Sessions/StereoSessionFactory.h>

#include <limits>
#include <cstring>

#include <boost/filesystem.hpp>
#include <boost/algorithm/string.hpp>
#include <boost/program_options.hpp>
#include <boost/algorithm/string/predicate.hpp>

// For parsing .json files
#include <nlohmann/json.hpp>

namespace fs = boost::filesystem;
namespace po = boost::program_options;
using json = nlohmann::json;

using namespace vw;
using namespace vw::camera;
using namespace vw::cartography;

// Solve for best fitting camera that projects given xyz locations at
// given pixels. If cam_weight > 0, try to constrain the camera height
// above datum at the value of cam_height.
// If camera center weight is given, use that to constrain the
// camera center, not just its height.
template <class CAM>
class CameraSolveLMA_Ht: public vw::math::LeastSquaresModelBase<CameraSolveLMA_Ht<CAM>> {
  std::vector<vw::Vector3> const& m_xyz;
  CAM m_camera_model;
  double m_cam_height, m_cam_weight, m_cam_ctr_weight;
  vw::cartography::Datum m_datum;
  Vector3 m_input_cam_ctr;
  
public:

  typedef vw::Vector<double>    result_type;   // pixel residuals
  typedef vw::Vector<double, 6> domain_type;   // camera parameters (camera center and axis angle)
  typedef vw::Matrix<double> jacobian_type;

  /// Instantiate the solver with a set of xyz to pixel pairs and a pinhole model
  CameraSolveLMA_Ht(std::vector<vw::Vector3> const& xyz,
                    CAM const& camera_model,
                    double cam_height, double cam_weight, double cam_ctr_weight,
                    vw::cartography::Datum const& datum):
    m_xyz(xyz),
    m_camera_model(camera_model), 
    m_cam_height(cam_height), m_cam_weight(cam_weight), m_cam_ctr_weight(cam_ctr_weight),
    m_datum(datum), m_input_cam_ctr(m_camera_model.camera_center(vw::Vector2())) {}

  /// Given the camera, project xyz into it
  inline result_type operator()(domain_type const& C) const {

    // Create the camera model
    CAM camera_model = m_camera_model;  // make a copy local to this function
    vector_to_camera(camera_model, C);  // update its parameters

    int xyz_len = m_xyz.size();
    size_t result_size = xyz_len * 2;
    if (m_cam_weight > 0)
      result_size += 1;
    else if (m_cam_ctr_weight > 0)
      result_size += 3; // penalize deviation from original camera center
     
    // See where the xyz coordinates project into the camera.
    result_type result;
    result.set_size(result_size);
    for (size_t i = 0; i < xyz_len; i++) {
      Vector2 pixel = camera_model.point_to_pixel(m_xyz[i]);
      result[2*i  ] = pixel[0];
      result[2*i+1] = pixel[1];
    }

    if (m_cam_weight > 0) {
      // Try to make the camera stay at given height
      Vector3 cam_ctr = subvector(C, 0, 3);
      Vector3 llh = m_datum.cartesian_to_geodetic(cam_ctr);
      result[2*xyz_len] = m_cam_weight*(llh[2] - m_cam_height);
    } else if (m_cam_ctr_weight > 0) {
      // Try to make the camera stay close to given center
      Vector3 cam_ctr = subvector(C, 0, 3);
      for (int it = 0; it < 3; it++) 
        result[2*xyz_len + it] = m_cam_ctr_weight*(m_input_cam_ctr[it] - cam_ctr[it]);
    }
    
    return result;
  }
}; // End class CameraSolveLMA_Ht

/// Find the best camera that fits the current GCP
void fit_camera_to_xyz_ht(bool parse_ecef,
			  Vector3 const& parsed_camera_center, // may not be known
                          Vector3 const& input_camera_center, // may not be known
			  std::string const& camera_type,
			  bool refine_camera, 
			  std::vector<Vector3> const& xyz_vec,
			  std::vector<double> const& pixel_values,
			  double cam_height, double cam_weight, double cam_ctr_weight,
			  vw::cartography::Datum const& datum,
			  bool verbose,
			  boost::shared_ptr<CameraModel> & out_cam){

  // Create fake points in space at given distance from this camera's
  // center and corresponding actual points on the ground.  Use 500
  // km, just some height not too far from actual satellite height.
  double ht = 500000.0; 
  int num_pts = pixel_values.size()/2;
  vw::Matrix<double> in, out;
  std::vector<vw::Vector2>  pixel_vec;
  in.set_size(3, num_pts);
  out.set_size(3, num_pts);
  for (int col = 0; col < in.cols(); col++) {
    if (input_camera_center != Vector3(0, 0, 0)) {
      // We know the camera center. Use that.
      ht = norm_2(xyz_vec[col] - input_camera_center);
    }
    Vector2 pix = Vector2(pixel_values[2*col], pixel_values[2*col+1]);
    Vector3 a = out_cam->camera_center(Vector2(0, 0)) + ht * out_cam->pixel_to_vector(pix);
    pixel_vec.push_back(pix);
    for (int row = 0; row < in.rows(); row++) {
      in(row, col)  = a[row];
      out(row, col) = xyz_vec[col][row];
    }
  }
  
  // Apply a transform to the camera so that the fake points are on top of the real points
  Matrix<double, 3, 3> rotation;
  Vector3 translation;
  double scale;
  find_3D_transform(in, out, rotation, translation, scale);
  if (camera_type == "opticalbar") {
    ((vw::camera::OpticalBarModel*)out_cam.get())->apply_transform(rotation,
								   translation, scale);
  } else {

    if (input_camera_center != Vector3(0, 0, 0)) {
      ((PinholeModel*)out_cam.get())->apply_transform(rotation, translation, scale);
    } else {
      // When we don't know the camera center, the logic based on fake points
      // can give junk results. Use instead the state-of-the-art OpenCV solver.
      // TODO(oalexan1): Need to consider using this solution also for OpticalBar
      // and even when we know the camera center.
      try {
        asp::findCameraPose(xyz_vec, pixel_vec, *(PinholeModel*)out_cam.get());
      } catch(std::exception const& e) {
        vw_out() << "Failed to find the camera pose using OpenCV. Falling back "
                  << "to ASP's internal approach. This is not as robust. "
                  << "Check your inputs and validate the produced camera.\n";
        // Fall back to previous logic
        ((PinholeModel*)out_cam.get())->apply_transform(rotation, translation, scale);
      }
    }
    
    if (parse_ecef) {
      // Overwrite the solved camera center with what is found from the
      // frame index file.
      ((PinholeModel*)out_cam.get())->set_camera_center(parsed_camera_center);
    }
  }
  
  // Print out some errors
  if (verbose) {
    vw_out() << "The error between the projection of each ground "
	     << "corner point into the coarse camera and its pixel value:\n";
    for (size_t corner_it = 0; corner_it < num_pts; corner_it++) {
      vw_out () << "Corner and error: ("
		<< pixel_values[2*corner_it] << ' ' << pixel_values[2*corner_it+1]
		<< ") " <<  norm_2(out_cam.get()->point_to_pixel(xyz_vec[corner_it]) -
				 Vector2( pixel_values[2*corner_it],
					  pixel_values[2*corner_it+1]))
		<< std::endl;
    }
  }

  Vector3 xyz0 = out_cam.get()->camera_center(vw::Vector2());

  // Solve a little optimization problem to make the points on the ground project
  // as much as possible exactly into the image corners.
  if (refine_camera) {
    Vector<double> out_vec; // must copy to this structure
    int residual_len = pixel_values.size();

    if (cam_weight > 0.0) 
      residual_len += 1; // for camera height residual
   else if (cam_ctr_weight > 0)
     residual_len += 3; // for camera center residual
 
    // Copy the image pixels
    out_vec.set_size(residual_len);
    for (size_t corner_it = 0; corner_it < pixel_values.size(); corner_it++) 
      out_vec[corner_it] = pixel_values[corner_it];

    // Use 0 for the remaining fields corresponding to camera height or 
    // camera center constraint
    for (int it = pixel_values.size(); it < residual_len; it++)
      out_vec[it] = 0.0;
      
    const double abs_tolerance  = 1e-24;
    const double rel_tolerance  = 1e-24;
    const int    max_iterations = 2000;
    int status = 0;
    Vector<double> final_params;
    Vector<double> seed;
      
    if (camera_type == "opticalbar") {
      CameraSolveLMA_Ht<vw::camera::OpticalBarModel>
	lma_model(xyz_vec, *((vw::camera::OpticalBarModel*)out_cam.get()),
		  cam_height, cam_weight, cam_ctr_weight, datum);
      camera_to_vector(*((vw::camera::OpticalBarModel*)out_cam.get()), seed);
      final_params = math::levenberg_marquardt(lma_model, seed, out_vec,
					       status, abs_tolerance, rel_tolerance,
					       max_iterations);
      vector_to_camera(*((vw::camera::OpticalBarModel*)out_cam.get()), final_params);
    } else {
      CameraSolveLMA_Ht<PinholeModel> lma_model(xyz_vec, *((PinholeModel*)out_cam.get()),
                                                cam_height, cam_weight, cam_ctr_weight, datum);
      camera_to_vector(*((PinholeModel*)out_cam.get()), seed);
      final_params = math::levenberg_marquardt(lma_model, seed, out_vec,
					       status, abs_tolerance, rel_tolerance,
					       max_iterations);
      vector_to_camera(*((PinholeModel*)out_cam.get()), final_params);
    }
    if (status < 1)
      vw_out() << "The Levenberg-Marquardt solver failed. Results may be inaccurate.\n";

    if (verbose) {
      vw_out() << "The error between the projection of each ground "
	       << "corner point into the refined camera and its pixel value:\n";
      for (size_t corner_it = 0; corner_it < num_pts; corner_it++) {
	vw_out () << "Corner and error: ("
		  << pixel_values[2*corner_it] << ' ' << pixel_values[2*corner_it+1]
		  << ") " <<  norm_2(out_cam.get()->point_to_pixel(xyz_vec[corner_it]) -
				     Vector2( pixel_values[2*corner_it],
					      pixel_values[2*corner_it+1]))
		  << std::endl;
      }
    }
    
  } // End camera refinement case
}

// Parse numbers or strings from a list where they are separated by commas or spaces.
template<class T>
void parse_values(std::string list, std::vector<T> & values){

  values.clear();

  // Replace commas with spaces
  std::string oldStr = ",", newStr = " ";
  size_t pos = 0;
  while((pos = list.find(oldStr, pos)) != std::string::npos){
    list.replace(pos, oldStr.length(), newStr);
    pos += newStr.length();
  }

  // Read the values one by one
  std::istringstream is(list);
  T val;
  while (is >> val)
    values.push_back(val);
}

struct Options : public vw::GdalWriteOptions {
  std::string image_file, camera_file, lon_lat_values_str, pixel_values_str, datum_str,
    reference_dem, frame_index, gcp_file, camera_type, sample_file, input_camera,
    stereo_session, bundle_adjust_prefix, parsed_cam_ctr_str, parsed_cam_quat_str;
  double focal_length, pixel_pitch, gcp_std, height_above_datum,
    cam_height, cam_weight, cam_ctr_weight;
  Vector2 optical_center;
  std::vector<double> lon_lat_values, pixel_values;
  bool refine_camera, parse_eci, parse_ecef, input_pinhole; 
  Options(): focal_length(-1), pixel_pitch(-1), gcp_std(1), height_above_datum(0), refine_camera(false), cam_height(0), cam_weight(0), cam_ctr_weight(0), input_pinhole(false) {}
};

void handle_arguments(int argc, char *argv[], Options& opt) {

  double nan = std::numeric_limits<double>::quiet_NaN();
  po::options_description general_options("");
  general_options.add_options()
    ("output-camera-file,o", po::value(&opt.camera_file), "Specify the output camera file with a .tsai extension.")
    ("camera-type", po::value(&opt.camera_type)->default_value("pinhole"), "Specify the camera type. Options are: pinhole (default) and opticalbar.")
    ("lon-lat-values", po::value(&opt.lon_lat_values_str)->default_value(""),
    "A (quoted) string listing numbers, separated by commas or spaces, "
    "having the longitude and latitude (alternating and in this "
    "order) of each image corner or some other list of pixels given "
    "by ``--pixel-values``. If the corners are used, they are traversed "
    "in the order (0, 0) (w, 0) (w, h), (0, h) where w and h are the "
     "image width and height.")
    ("pixel-values", po::value(&opt.pixel_values_str)->default_value(""), "A (quoted) string listing numbers, separated by commas or spaces, having the column and row (alternating and in this order) of each pixel in the raw image at which the longitude and latitude is known and given by --lon-lat-values. By default this is empty, and will be populated by the image corners traversed as mentioned at the earlier option.")
    ("reference-dem", po::value(&opt.reference_dem)->default_value(""),
     "Use this DEM to infer the heights above datum of the image corners.")
    ("datum", po::value(&opt.datum_str)->default_value(""),
     "Use this datum to interpret the longitude and latitude, unless a DEM is given. Options: WGS_1984, D_MOON (1,737,400 meters), D_MARS (3,396,190 meters), MOLA (3,396,000 meters), NAD83, WGS72, and NAD27. Also accepted: Earth (=WGS_1984), Mars (=D_MARS), Moon (=D_MOON).")
    ("height-above-datum", po::value(&opt.height_above_datum)->default_value(0),
     "Assume this height above datum in meters for the image corners unless read from the DEM.")
    ("sample-file", po::value(&opt.sample_file)->default_value(""), 
     "Read in the camera parameters from the example camera file.  Required for opticalbar type.")
    ("focal-length", po::value(&opt.focal_length)->default_value(0),
     "The camera focal length.")
    ("optical-center", po::value(&opt.optical_center)->default_value(Vector2(nan, nan),"NaN NaN"),
     "The camera optical center. If not specified for pinhole cameras, it will be set to image center (half of image dimensions) times the pixel pitch. The optical bar camera always uses the image center.")
    ("pixel-pitch", po::value(&opt.pixel_pitch)->default_value(0),
     "The pixel pitch.")
    ("refine-camera", po::bool_switch(&opt.refine_camera)->default_value(false),
     "After a rough initial camera is obtained, refine it using least squares.")
    ("frame-index", po::value(&opt.frame_index)->default_value(""),
     "A file used to look up the longitude and latitude of image corners based on the image name, in the format provided by the SkySat video product.")
    ("gcp-file", po::value(&opt.gcp_file)->default_value(""),
     "If provided, save the image corner coordinates and heights in the GCP format to this file.")
    ("gcp-std", po::value(&opt.gcp_std)->default_value(1),
     "The standard deviation for each GCP pixel, if saving a GCP file. A smaller value suggests a more reliable measurement, hence will be given more weight.")
    ("cam-height", po::value(&opt.cam_height)->default_value(0),
     "If both this and --cam-weight are positive, enforce that the output camera is at this height above datum. For SkySat, if not set, read this from the frame index. Highly experimental.")
    ("cam-weight", po::value(&opt.cam_weight)->default_value(0),
     "If positive, try to enforce the option --cam-height with this weight (bigger weight means try harder to enforce).")
    ("cam-ctr-weight", po::value(&opt.cam_ctr_weight)->default_value(0),
     "If positive, try to enforce that during camera refinement the camera center stays close to the initial value (bigger weight means try harder to enforce this; a value like 1000.0 is good enough).")
    ("parse-eci", po::bool_switch(&opt.parse_eci)->default_value(false),
     "Create cameras based on ECI positions and orientations (not working).")
    ("parse-ecef", po::bool_switch(&opt.parse_ecef)->default_value(false),
     "Create cameras based on ECEF position (but not orientation).")
    ("input-camera", po::value(&opt.input_camera)->default_value(""),
     "Create the output pinhole camera approximating this camera. If with a "
     "_pinhole.json suffix, read it verbatim, with no refinements or "
     "taking into account other input options.")
    ("session-type,t",   po::value(&opt.stereo_session)->default_value(""),
     "Select the input camera model type. Normally this is auto-detected, but may need to be specified if the input camera model is in XML format. See the doc for options.")
    ("bundle-adjust-prefix", po::value(&opt.bundle_adjust_prefix),
     "Use the camera adjustment obtained by previously running bundle_adjust "
     "when providing an input camera.");
  
  general_options.add(vw::GdalWriteOptionsDescription(opt));

  po::options_description positional("");
  positional.add_options()
    ("image-file", po::value(&opt.image_file));

  po::positional_options_description positional_desc;
  positional_desc.add("image-file",1);

  std::string usage("[options] <image-file> -o <camera-file>");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                            positional, positional_desc, usage,
                            allow_unregistered, unregistered);

  if (opt.image_file.empty())
    vw_throw( ArgumentErr() << "Missing the input image.\n"
              << usage << general_options );

  if (opt.camera_file.empty())
    vw_throw( ArgumentErr() << "Missing the output camera file name.\n"
              << usage << general_options );

  boost::to_lower(opt.camera_type);
  
  if (opt.camera_type != "pinhole" && opt.camera_type != "opticalbar")
    vw_throw( ArgumentErr() << "Only pinhole and opticalbar cameras are supported.\n");
  
  if ((opt.camera_type == "opticalbar") && (opt.sample_file == ""))
    vw_throw( ArgumentErr() << "opticalbar type must use a sample camera file.\n"
              << usage << general_options );

  std::string ext = get_extension(opt.camera_file);
  if (ext != ".tsai") 
    vw_throw( ArgumentErr() << "The output camera file must end with .tsai.\n"
              << usage << general_options );

  opt.input_pinhole = boost::algorithm::ends_with(opt.input_camera, "_pinhole.json");
  
  // If we cannot read the data from a DEM, must specify a lot of things.
  if (!opt.input_pinhole && opt.reference_dem.empty() && opt.datum_str.empty())
    vw_throw( ArgumentErr() << "Must provide either a reference DEM or a datum.\n"
              << usage << general_options );

  if (opt.gcp_std <= 0) 
    vw_throw( ArgumentErr() << "The GCP standard deviation must be positive.\n"
              << usage << general_options );

  if (!opt.input_pinhole && opt.frame_index != "" && opt.lon_lat_values_str != "") 
    vw_throw( ArgumentErr() << "Cannot specify both the frame index file "
	      << "and the lon-lat corners.\n"
              << usage << general_options );

  if (opt.cam_weight > 0 && opt.cam_ctr_weight > 0)
    vw::vw_throw(vw::ArgumentErr() << "Cannot enforce the camera center constraint and camera height constraint at the same time.\n");

  if (!opt.input_pinhole && opt.frame_index != "") {
    // Parse the frame index to extract opt.lon_lat_values_str.
    // Look for a line having this image, and search for "POLYGON" followed by spaces and "((".
    boost::filesystem::path p(opt.image_file); 
    std::string image_base = p.stem().string(); // strip the directory name and suffix
    std::ifstream file( opt.frame_index.c_str() );
    std::string line;
    std::string beg1 = "POLYGON";
    std::string beg2 = "((";
    std::string end = "))";
    while (getline(file, line, '\n')) {
      if (line.find(image_base) != std::string::npos) {
        // Find POLYGON first.
        int beg_pos = line.find(beg1);
        if (beg_pos == std::string::npos)
          vw_throw( ArgumentErr() << "Cannot find " << beg1 << " in line: " << line << ".\n");
        beg_pos += beg1.size();

        // Move forward skipping any spaces until finding "(("
        beg_pos = line.find(beg2, beg_pos);
        if (beg_pos == std::string::npos)
          vw_throw( ArgumentErr() << "Cannot find " << beg2 << " in line: " << line << ".\n");
        beg_pos += beg2.size();

        // Find "))"
        int end_pos = line.find(end, beg_pos);
        if (end_pos == std::string::npos)
          vw_throw( ArgumentErr() << "Cannot find " << end << " in line: " << line << ".\n");
        opt.lon_lat_values_str = line.substr(beg_pos, end_pos - beg_pos);
        vw_out() << "Parsed the lon-lat corner values: " << opt.lon_lat_values_str
		 << std::endl;

	if (opt.parse_eci && opt.parse_ecef)
	  vw_throw( ArgumentErr() << "Cannot parse both ECI end ECEF at the same time.\n");
	
	// Also parse the camera height constraint, unless manually specified
	if (opt.cam_weight > 0 || opt.parse_eci || opt.parse_ecef) {
	  std::vector<std::string> vals;
	  parse_values<std::string>(line, vals);
	  
	  if (vals.size() < 12) 
	    vw_throw( ArgumentErr() << "Could not parse 12 values from: " << line << ".\n");

	  // Extract the ECI or ECEF coordinates of camera
	  // center. Keep them as string until we can convert to
	  // height above datum.
	  
	  if (opt.parse_eci) {
	    std::string x = vals[5];
	    std::string y = vals[6];
	    std::string z = vals[7];
	    opt.parsed_cam_ctr_str = x + " " + y + " " + z;
	    vw_out() << "Parsed the ECI camera center in km: "
		     << opt.parsed_cam_ctr_str <<".\n";
	    
	    std::string q0 = vals[8];
	    std::string q1 = vals[9];
	    std::string q2 = vals[10];
	    std::string q3 = vals[11];
	    opt.parsed_cam_quat_str = q0 + " " + q1 + " " + q2 + " " + q3;
	    vw_out() << "Parsed the ECI quaternion: "
		     << opt.parsed_cam_quat_str <<".\n";
	  }
	  
	  if (opt.parse_ecef) {
	    if (vals.size() < 19) 
	      vw_throw( ArgumentErr() << "Could not parse 19 values from: " << line << ".\n");
	    
	    std::string x = vals[12];
	    std::string y = vals[13];
	    std::string z = vals[14];
	    opt.parsed_cam_ctr_str = x + " " + y + " " + z;
	    vw_out() << "Parsed the ECEF camera center in km: "
		     << opt.parsed_cam_ctr_str <<".\n";
	    
	    std::string q0 = vals[15];
	    std::string q1 = vals[16];
	    std::string q2 = vals[17];
	    std::string q3 = vals[18];
	    opt.parsed_cam_quat_str = q0 + " " + q1 + " " + q2 + " " + q3;
	    vw_out() << "Parsed the ECEF quaternion: "
		     << opt.parsed_cam_quat_str <<".\n";
	  }
	  
	}
	
        break;
      }
    }
    if (opt.lon_lat_values_str == "")
      vw_throw( ArgumentErr() << "Could not parse the entry for " << image_base
                << " in file: " << opt.frame_index << ".\n");
  }
    
  // Parse the pixel values
  parse_values<double>(opt.pixel_values_str, opt.pixel_values);

  // If none were provided, use the image corners
  if (!opt.input_pinhole && opt.pixel_values.empty()) {
    DiskImageView<float> img(opt.image_file);
    int wid = img.cols(), hgt = img.rows();
    if (wid <= 0 || hgt <= 0) 
      vw_throw( ArgumentErr() << "Could not read an image with positive dimensions from: "
		<< opt.image_file << ".\n");
    
    // populate the corners
    double arr[] = {0.0, 0.0, (double)wid, 0.0, (double)wid, (double)hgt, 0.0, (double)hgt};
    for (size_t it  = 0; it < sizeof(arr)/sizeof(double); it++) 
      opt.pixel_values.push_back(arr[it]);

    // Add inner points for robustness
    if (opt.input_camera != "") {
      double b = 0.25, e = 0.75;
      double arr[] = {b*wid, b*hgt, e*wid, b*hgt, e*wid, e*hgt, b*wid, e*hgt};
      for (size_t it  = 0; it < sizeof(arr)/sizeof(double); it++) 
	opt.pixel_values.push_back(arr[it]);
    }
    
  }
    
  // Parse the lon-lat values
  if (!opt.input_pinhole && opt.input_camera == "") {
    parse_values<double>(opt.lon_lat_values_str, opt.lon_lat_values);
    // Bug fix for some frame_index files repeating the first point at the end
    int len = opt.lon_lat_values.size();
    if (opt.frame_index != "" && opt.lon_lat_values.size() == opt.pixel_values.size() + 2 &&
        len >= 2 && opt.lon_lat_values[0] == opt.lon_lat_values[len - 2] &&
        opt.lon_lat_values[1] == opt.lon_lat_values[len - 1]) {
      opt.lon_lat_values.pop_back();
      opt.lon_lat_values.pop_back();
    }
  }
  
  // Note that optical center can be negative (for some SkySat products).
  if (!opt.input_pinhole &&
      opt.sample_file == "" &&
      (opt.focal_length <= 0 || opt.pixel_pitch <= 0))
    vw_throw( ArgumentErr() << "Must provide positive focal length"
              << "and pixel pitch values OR a sample file.\n");

  if ((opt.parse_eci || opt.parse_ecef) && opt.camera_type == "opticalbar") 
    vw_throw( ArgumentErr() << "Cannot parse ECI/ECEF data for an optical bar camera.\n");
  
  // Create the output directory
  vw::create_out_dir(opt.camera_file);

} // End function handle_arguments

// Form a camera based on info the user provided
void manufacture_cam(Options const& opt, int wid, int hgt,
		     boost::shared_ptr<CameraModel> & out_cam){

  if (opt.camera_type == "opticalbar") {
    boost::shared_ptr<vw::camera::OpticalBarModel> opticalbar_cam;
    opticalbar_cam.reset(new vw::camera::OpticalBarModel(opt.sample_file));
    // Make sure the image size matches the input image file.
    // TODO(oalexan1): This looks fishy if the pitch is not 1.
    opticalbar_cam->set_image_size(Vector2i(wid, hgt));
    opticalbar_cam->set_optical_center(Vector2(wid/2.0, hgt/2.0));
    out_cam = opticalbar_cam;
  } else {
    boost::shared_ptr<PinholeModel> pinhole_cam;
    if (opt.sample_file != "") {
      // Use the initial guess from file
      pinhole_cam.reset(new PinholeModel(opt.sample_file));
    } else {
      // Use the intrinsics from the command line. Use trivial rotation and translation.
      Vector3 ctr(0, 0, 0);
      Matrix<double, 3, 3> rotation;
      rotation.set_identity();
      // When the user does not set the optical center, use the image center times pixel pitch
      Vector2 opt_ctr = opt.optical_center;
      if (std::isnan(opt_ctr[0]) || std::isnan(opt_ctr[1]))
        opt_ctr = Vector2(opt.pixel_pitch * wid/2.0, opt.pixel_pitch * hgt/2.0);

      pinhole_cam.reset(new PinholeModel(ctr, rotation, opt.focal_length, opt.focal_length,
					 opt_ctr[0], opt_ctr[1],
					 NULL, opt.pixel_pitch));
    }
    out_cam = pinhole_cam;
  }
}

// TODO: Wipe this logic and use RayDEMIntersectionLMA from VW.
// That one is also terrible code which needs to be replaced with a
// proper root-finding algorithm
// and use it. And this code should be moved to VW.
// https://github.com/NeoGeographyToolkit/StereoPipeline/issues/267
namespace vw {
  namespace cartography {

  // Define an LMA model to solve for a DEM intersecting a ray. The
  // variable of optimization is position on the ray. The cost
  // function is difference between datum height and DEM height at
  // current point on the ray.
  template <class DEMImageT>
  class RayDEMIntersectionLMA2 : public math::LeastSquaresModelBase<RayDEMIntersectionLMA2<DEMImageT>> {

    // TODO: Why does this use EdgeExtension if Helper() restricts access to the bounds?
    InterpolationView<EdgeExtensionView<DEMImageT, ConstantEdgeExtension>,
                      BilinearInterpolation> m_dem;
    GeoReference const& m_georef; // alias
    Vector3      m_camera_ctr;
    Vector3      m_camera_vec;
    bool         m_treat_nodata_as_zero;

    /// Provide safe interaction with DEMs that are scalar
    /// - If m_dem(x,y) is in bounds, return the interpolated value.
    /// - Otherwise return 0 or big_val()
    template <class PixelT>
    typename boost::enable_if< IsScalar<PixelT>, double >::type
    inline Helper( double x, double y ) const {
      if ( (0 <= x) && (x <= m_dem.cols() - 1) && // for interpolation
           (0 <= y) && (y <= m_dem.rows() - 1)) {
        PixelT val = m_dem(x, y);
        if (is_valid(val)) return val;
      }
      if (m_treat_nodata_as_zero) return 0;
      return big_val();
    }

    /// Provide safe interaction with DEMs that are compound
    template <class PixelT>
    typename boost::enable_if< IsCompound<PixelT>, double>::type
    inline Helper( double x, double y ) const {
      if ( (0 <= x) && (x <= m_dem.cols() - 1) && // for interpolation
           (0 <= y) && (y <= m_dem.rows() - 1) ){
        PixelT val = m_dem(x, y);
        if (is_valid(val)) return val[0];
      }
      if (m_treat_nodata_as_zero) return 0;
      return big_val();
    }

  public:
    typedef Vector<double, 1> result_type;
    typedef Vector<double, 1> domain_type;
    typedef Matrix<double>    jacobian_type; ///< Jacobian form. Auto.

    /// Return a very large error to penalize locations that fall off the edge of the DEM.
    inline double big_val() const {
      // Don't make this too big as in the LMA algorithm it may get squared and may cause overflow.
      return 1.0e+50;
    }

    /// Constructor
    RayDEMIntersectionLMA2(ImageViewBase<DEMImageT> const& dem_image,
                           GeoReference const& georef,
                           Vector3 const& camera_ctr,
                           Vector3 const& camera_vec,
                           bool treat_nodata_as_zero):
      m_dem(interpolate(dem_image)), m_georef(georef),
      m_camera_ctr(camera_ctr), m_camera_vec(camera_vec),
      m_treat_nodata_as_zero(treat_nodata_as_zero) {}

    /// Evaluator. See description above.
    inline result_type operator()( domain_type const& len ) const {
      // The proposed intersection point
      Vector3 xyz = m_camera_ctr + len[0]*m_camera_vec;

      // Convert to geodetic coordinates, then to DEM pixel coordinates
      Vector3 llh = m_georef.datum().cartesian_to_geodetic( xyz );
      Vector2 pix = m_georef.lonlat_to_pixel( Vector2( llh.x(), llh.y() ) );
      
      // Return a measure of the elevation difference between the DEM and the guess
      // at its current location.
      result_type result;
      result[0] = Helper<typename DEMImageT::pixel_type >(pix.x(),pix.y()) - llh[2];
      return result;
    }
  };

    
  // Intersect the ray going from the given camera pixel with a DEM.
  // The return value is a Cartesian point. If the ray goes through a
  // hole in the DEM where there is no data, we return no-intersection
  // or intersection with the datum, depending on whether the variable
  // treat_nodata_as_zero is false or true.
  template <class DEMImageT>
  Vector3 camera_pixel_to_dem_xyz2(Vector3 const& camera_ctr, Vector3 const& camera_vec,
                                  ImageViewBase<DEMImageT> const& dem_image,
                                  GeoReference const& georef,
                                  bool treat_nodata_as_zero,
                                  bool & has_intersection,
                                  double height_error_tol = 1e-1,  // error in DEM height
                                  double max_abs_tol      = 1e-14, // abs cost fun change b/w iters
                                  double max_rel_tol      = 1e-14,
                                  int num_max_iter        = 100,
                                  Vector3 xyz_guess       = Vector3()){

    // This is a very fragile function and things can easily go wrong. 
    try {
      has_intersection = false;
      RayDEMIntersectionLMA2<DEMImageT> model(dem_image, georef, camera_ctr,
                                             camera_vec, treat_nodata_as_zero);

      Vector3 xyz;
      if ( xyz_guess == Vector3() ){ // If no guess provided
        // Intersect the ray with the datum, this is a good initial guess.
        xyz = datum_intersection(georef.datum(), camera_ctr, camera_vec);

        if ( xyz == Vector3() ) { // If we failed to intersect the datum, give up!
          has_intersection = false;
          return Vector3();
        }
      }else{ // User provided guess
        xyz = xyz_guess;
      }

      // Length along the ray from camera center to datum intersection point
      Vector<double, 1> base_len, len;
      double smallest_error_pos = std::numeric_limits<double>::max();
      double best_len_pos = std::numeric_limits<double>::max();
      double smallest_error_neg = std::numeric_limits<double>::max();
      double best_len_neg = std::numeric_limits<double>::max();
      bool success_pos = false, success_neg = false;
      
      // If the ray intersects the datum at a point which does not
      // correspond to a valid location in the DEM, wiggle that point
      // along the ray until hopefully it does. Store the value that
      // is closest to where that ray will intersect the DEM. Once
      // that value is located, it is helpful to repeat this logic one
      // more time, this time around the best guess found so far.
      // Hence two outer passes. The value xyz is updated at each
      // pass. The idea here is that the closer one gets to the true
      // solution, the likelier the LM solver will converge.
      for (int outer_pass = 0; outer_pass <= 0; outer_pass++){
	
	base_len[0] = norm_2(xyz - camera_ctr);
      
	const double radius     = norm_2(xyz); // Radius from XYZ coordinate center
	const int    ITER_LIMIT = 10; // There are two solver attempts per iteration
	const double small      = radius*0.02/( 1 << (ITER_LIMIT-1) ); // Wiggle
	for (int i = 0; i <= ITER_LIMIT; i++){
	  // Gradually expand delta until on final iteration it is == radius*0.02
	  double delta = 0;
	  if (i > 0)
	    delta = small*( 1 << (i-1) );

	  for (int k = -1; k <= 1; k += 2){ // For k==-1, k==1
	    len[0] = base_len[0] + k*delta; // Ray guess length +/- 2% planetary radius
	    // Use our model to compute the height diff at this length

	    Vector<double, 1> height_diff = model(len);
	  
	    if ( std::abs(height_diff[0]) < (model.big_val()/10.0) ){
	      has_intersection = true;
	    }else{
	      continue;
	    }
	    //if (i == 0) break; // When k*delta==0, no reason to do both + and -!

	    if (height_diff[0] < 0 && std::abs(height_diff[0]) < smallest_error_neg){
	      
	      smallest_error_neg = std::abs(height_diff[0]);
	      best_len_neg = len[0];
	      xyz = camera_ctr + best_len_neg*camera_vec; // broken!!!
	      success_neg = true;
	    }else{
	    }

	    if (height_diff[0] >=0 && std::abs(height_diff[0]) < smallest_error_pos){
	      
	      smallest_error_pos = std::abs(height_diff[0]);
	      best_len_pos = len[0];
	      success_pos = true;
	      xyz = camera_ctr + best_len_pos*camera_vec; // broken!!!
	    }else{
	    }

	    
	  } // End k loop
	  if (has_intersection) {
	    // break;
	  }
	} // End i loop
      
	// Failed to compute an intersection in the hard coded iteration limit!
	if ( !has_intersection ) {
	  return Vector3();
	}
      }

      // Refining the intersection using Levenberg-Marquardt
      // - This will actually use the L-M solver to play around with the len
      //   value to minimize the height difference from the DEM.
      int status = 0;
      Vector<double, 1> observation;
      observation[0] = 0;
      Vector<double, 1> dem_height_neg;
      dem_height_neg[0] = std::numeric_limits<double>::max();
      Vector<double, 1> final_len_neg;
      if (success_neg) {
	len[0] = best_len_neg;
	final_len_neg = math::levenberg_marquardt(model, len, observation, status,
                                      max_abs_tol, max_rel_tol,
						  num_max_iter);
	dem_height_neg = model(final_len_neg);
	
	if (status < 0) 
	  success_neg = false;
      }
      

      status = 0;
      observation[0] = 0;
      len[0] = best_len_pos;
      Vector<double, 1> final_len_pos;
      Vector<double, 1> dem_height_pos;
      dem_height_pos[0] = std::numeric_limits<double>::max();
      if (success_pos) {
	final_len_pos = math::levenberg_marquardt(model, len, observation, status,
				    max_abs_tol, max_rel_tol,
				    num_max_iter
				    );
	dem_height_pos = model(final_len_pos);
	if (status < 0) 
	  success_pos = false;
      }

      Vector<double, 1> dem_height;
      if (success_pos && std::abs(dem_height_pos[0]) <= std::abs(dem_height_neg[0])) {
	dem_height = dem_height_pos;
	len = final_len_pos;
      }else if (success_neg && std::abs(dem_height_neg[0]) <= std::abs(dem_height_pos[0])){
	dem_height = dem_height_neg;
	len = final_len_neg;
      }
      
      vw_out() << "Height error: " << dem_height << std::endl;
      
      if (!success_pos && !success_neg) 
	status = -1;
      
      if ( (status < 0) || (std::abs(dem_height[0]) > height_error_tol) ){
        has_intersection = false;
        return Vector3();
      }

      has_intersection = true;
      xyz = camera_ctr + len[0]*camera_vec;
      return xyz;
    }catch(...){
      has_intersection = false;
    }
    return Vector3();
  }

}
}

// Trace rays from pixel corners to DEM to see where they intersect the DEM
void extract_lon_lat_cam_ctr_from_camera(Options & opt,
                                         ImageViewRef<PixelMask<float>> const& interp_dem,
				 GeoReference const& geo,
                                 std::vector<double> & cam_heights, vw::Vector3 & cam_ctr) {

  cam_heights.clear();
  cam_ctr = Vector3(0, 0, 0);
  
  // Need this to be able to load adjusted camera models. That will happen
  // in the stereo session.
  asp::stereo_settings().bundle_adjust_prefix = opt.bundle_adjust_prefix;
  
  std::string out_prefix;
  typedef boost::scoped_ptr<asp::StereoSession> SessionPtr;
  SessionPtr session(asp::StereoSessionFactory::create(opt.stereo_session, // may change
						       opt,
						       opt.image_file, opt.image_file,
						       opt.input_camera, opt.input_camera,
						       out_prefix));

  boost::shared_ptr<CameraModel> camera_model = session->camera_model(opt.image_file,
								      opt.input_camera);

  // Store here pixel values for the rays emanating from the pixels at
  // which we could intersect with the DEM.
  std::vector<double> good_pixel_values;
  
  int num_points = opt.pixel_values.size()/2;
  opt.lon_lat_values.reserve(2*num_points);
  opt.lon_lat_values.clear();

  // Estimate camera center
  std::vector<vw::Vector3> ctrs, dirs;
  
  for (int it = 0; it < num_points; it++){

    Vector2 pix(opt.pixel_values[2*it], opt.pixel_values[2*it+1]);

    Vector3 camera_ctr = camera_model->camera_center(pix);
    Vector3 camera_vec = camera_model->pixel_to_vector(pix);

    bool treat_nodata_as_zero = false;
    bool has_intersection = false;
    double height_error_tol = 1.0; // error in DEM height
    
    double max_abs_tol = 1e-20;
    double max_rel_tol      = 1e-20;
    int num_max_iter        = 1000;
    Vector3 xyz_guess       = Vector3();

    Vector3 xyz = camera_pixel_to_dem_xyz2(camera_ctr, camera_vec,  
                                           interp_dem, geo, treat_nodata_as_zero,
					   has_intersection, height_error_tol,
					   max_abs_tol, max_rel_tol, num_max_iter, xyz_guess);
    
    if (xyz == Vector3() || !has_intersection){
      vw_out() << "Could not intersect the DEM with a ray coming "
	       << "from the camera at pixel: " << pix << ". Skipping it.\n";
      continue;
    }

    ctrs.push_back(camera_ctr);
    dirs.push_back(camera_vec);
    
    Vector3 llh = geo.datum().cartesian_to_geodetic(xyz);
    opt.lon_lat_values.push_back(llh[0]);
    opt.lon_lat_values.push_back(llh[1]);
    good_pixel_values.push_back(opt.pixel_values[2*it]);
    good_pixel_values.push_back(opt.pixel_values[2*it+1]);
    cam_heights.push_back(llh[2]); // will use it later
  }

  if (good_pixel_values.size() < 6) {
    vw_throw( ArgumentErr() << "Successful intersection happened for less than "
	      << "3 pixels. Will not be able to create a camera. Consider checking "
	      << "your inputs, or passing different pixels in --pixel-values. DEM: "
	      << opt.reference_dem << ".\n");
  }

  // Estimate camera center by triangulating back to the camera. This is necessary
  // for RPC, which does not store a camera center
  int num = 0;
  for (size_t it1 = 0; it1 < ctrs.size(); it1++) {
    for (size_t it2 = it1 + 1; it2 < ctrs.size(); it2++) {
      vw::Vector3 err;
      vw::Vector3 pt = vw::stereo::triangulate_pair(dirs[it1], ctrs[it1],
                                                    dirs[it2], ctrs[it2], err);
      if (pt != Vector3(0, 0, 0)) {
        cam_ctr += pt;
        num += 1;
      }
    }
  }
  if (num > 0) 
    cam_ctr = cam_ctr / num;
  
  // Update with the values at which we were successful
  opt.pixel_values = good_pixel_values;
}

vw::Matrix<double> vec2matrix(int rows, int cols, std::vector<double> const& vals) {
  int len = vals.size();
  if (len != rows * cols) 
    vw::vw_throw(vw::ArgumentErr() << "Size mis-match.\n");

  vw::Matrix<double> M;
  M.set_size(rows, cols);

  int count = 0;
  for (int row = 0; row < rows; row++) {
    for (int col = 0; col < cols; col++) {
      M(row, col) = vals[count];
      count++;
    }
  }
  return M;
}

// Read a matrix in json format. This will throw an error if the json object
// does not have the expected data.
vw::Matrix<double> json_mat(json const& j, int rows, int cols) {

  vw::Matrix<double> M;
  M.set_size(rows, cols);
  for (int row = 0; row < rows; row++) {
    for (int col = 0; col < cols; col++) {
      M(row, col) = j[row][col].get<double>();
    }
  }
  return M;
}

// Create a pinhole camera using user-specified options.
void form_pinhole_camera(Options & opt, vw::cartography::Datum & datum,
                         boost::shared_ptr<CameraModel> & out_cam) {

  GeoReference geo;
  ImageView<float> dem;
  float nodata_value = -std::numeric_limits<float>::max(); 
  bool has_dem = false;
  if (opt.reference_dem != "") {
    dem = DiskImageView<float>(opt.reference_dem);
    bool ans = read_georeference(geo, opt.reference_dem);
    if (!ans) 
      vw_throw( ArgumentErr() << "Could not read the georeference from dem: "
                << opt.reference_dem << ".\n");

    datum = geo.datum(); // Read this in for completeness
    has_dem = true;
    vw::read_nodata_val(opt.reference_dem, nodata_value);
    vw_out() << "Using nodata value: " << nodata_value << std::endl;
  }else{
    datum = vw::cartography::Datum(opt.datum_str); 
    vw_out() << "No reference DEM provided. Will use a height of "
             << opt.height_above_datum << " above the datum:\n" 
             << datum << std::endl;
  }

  // Prepare the DEM for interpolation
  ImageViewRef<PixelMask<float>> interp_dem
    = interpolate(create_mask(dem, nodata_value),
                  BilinearInterpolation(), ZeroEdgeExtension());

  // If we have camera center in ECI or ECEF coordinates in km, convert
  // it to meters, then find the height above datum.
  Vector3 parsed_cam_ctr(0, 0, 0);
  if (opt.parsed_cam_ctr_str != "") {
    std::vector<double> vals;
    parse_values<double>(opt.parsed_cam_ctr_str, vals);
    if (vals.size() != 3) 
      vw_throw( ArgumentErr() << "Could not parse 3 values from: "
                << opt.parsed_cam_ctr_str << ".\n");

    parsed_cam_ctr = Vector3(vals[0], vals[1], vals[2]);
    parsed_cam_ctr *= 1000.0;  // convert to meters
    vw_out() << "Parsed camera center (meters): " << parsed_cam_ctr << "\n";

    Vector3 llh = datum.cartesian_to_geodetic(parsed_cam_ctr);
      
    // If parsed_cam_ctr is in ECI coordinates, the lon and lat won't be accurate
    // but the height will be.
    if (opt.cam_weight > 0) 
      opt.cam_height = llh[2];
  }
    
  vw::Quat parsed_cam_quat;
  if (opt.parsed_cam_quat_str != "") {
    std::vector<double> vals;
    parse_values<double>(opt.parsed_cam_quat_str, vals);
    if (vals.size() != 4) 
      vw_throw( ArgumentErr() << "Could not parse 4 values from: "
                << opt.parsed_cam_quat_str << ".\n");

    parsed_cam_quat = vw::Quat(vals[0], vals[1], vals[2], vals[3]);
    vw_out() << "Parsed camera quaternion: " << parsed_cam_quat << "\n";
  }
    
  if (opt.cam_weight > 0) {
    vw_out() << "Will attempt to find a camera center height above datum of "
             << opt.cam_height
             << " meters with a weight strength of " << opt.cam_weight << ".\n";
  }
  if (opt.cam_ctr_weight > 0 && opt.refine_camera)  
    vw_out() << "Will try to have the camera center change little during camera refinement.\n"; 

  Vector3 input_cam_ctr(0, 0, 0); // estimated camera center from input camera
  std::vector<double> cam_heights;
  if (opt.input_camera != ""){
    // Extract lon and lat from tracing rays from the camera to the ground.
    // This can modify opt.pixel_values. Also calc the camera center.
    extract_lon_lat_cam_ctr_from_camera(opt, create_mask(dem, nodata_value), geo, cam_heights,
                                        input_cam_ctr);
  }

  // Overwrite the estimated center with what is parsed from vendor's data,
  // if this data exists.
  if (opt.parse_ecef && parsed_cam_ctr != Vector3())
    input_cam_ctr = parsed_cam_ctr;
    
  if (opt.lon_lat_values.size() < 3) 
    vw_throw( ArgumentErr() << "Expecting at least three longitude-latitude pairs.\n");

  if (opt.lon_lat_values.size() != opt.pixel_values.size()){
    vw_throw( ArgumentErr()
              << "The number of lon-lat pairs must equal the number of pixel pairs.\n");
  }

  size_t num_lon_lat_pairs = opt.lon_lat_values.size()/2;
    
  Vector2 pix;
  Vector3 llh, xyz;
  std::vector<Vector3> xyz_vec;

  // If to write a gcp file
  std::ostringstream gcp;
  gcp.precision(17);
  bool write_gcp = (opt.gcp_file != "");

  // TODO(oalexan1): Make this into a function
  for (size_t corner_it = 0; corner_it < num_lon_lat_pairs; corner_it++) {

    // Get the height from the DEM if possible
    llh[0] = opt.lon_lat_values[2*corner_it+0];
    llh[1] = opt.lon_lat_values[2*corner_it+1];

    if (llh[1] < -90 || llh[1] > 90) 
      vw_throw( ArgumentErr() << "Detected a latitude out of bounds. "
                << "Perhaps the longitude and latitude are reversed?\n");

    double height = opt.height_above_datum; 
    if (opt.input_camera != ""){
      height = cam_heights[corner_it]; // already computed
    } else {
      if (has_dem) {
        bool success = false;
        pix = geo.lonlat_to_pixel(subvector(llh, 0, 2));
        int len =  BilinearInterpolation::pixel_buffer;
        if (pix[0] >= 0 && pix[0] <= interp_dem.cols() - 1 - len &&
            pix[1] >= 0 && pix[1] <= interp_dem.rows() - 1 - len) {
          PixelMask<float> masked_height = interp_dem(pix[0], pix[1]);
          if (is_valid(masked_height)) {
            height = masked_height.child();
            success = true;
          }
        }
        if (!success) 
          vw_out() << "Could not determine a valid height value at lon-lat: "
                   << llh[0] << ' ' << llh[1] << ". Will use a height of " << height << ".\n";
      }
    }
      
    llh[2] = height;
    //vw_out() << "Lon-lat-height for corner ("
    //         << opt.pixel_values[2*corner_it] << ", " << opt.pixel_values[2*corner_it+1]
    //         << ") is "
    //         << llh[0] << ", " << llh[1] << ", " << llh[2] << std::endl;

    xyz = datum.geodetic_to_cartesian(llh);
    xyz_vec.push_back(xyz);

    if (write_gcp)
      gcp << corner_it << ' ' << llh[1] << ' ' << llh[0] << ' ' << llh[2] << ' '
          << 1 << ' ' << 1 << ' ' << 1 << ' ' << opt.image_file << ' '
          << opt.pixel_values[2*corner_it] << ' ' << opt.pixel_values[2*corner_it+1] << ' '
          << opt.gcp_std << ' ' << opt.gcp_std << std::endl;
  } // End loop through lon-lat pairs

  if (write_gcp) {
    vw_out() << "Writing: " << opt.gcp_file << std::endl;
    std::ofstream fs(opt.gcp_file.c_str());
    fs << gcp.str();
    fs.close();
  }
    
  // Form a camera based on info the user provided
  DiskImageView<float> img(opt.image_file);
  int wid = img.cols(), hgt = img.rows();
  if (wid <= 0 || hgt <= 0) 
    vw_throw( ArgumentErr() << "Could not read an image with positive dimensions from: "
              << opt.image_file << ".\n");
  manufacture_cam(opt, wid, hgt, out_cam);

  // Transform it and optionally refine it
  bool verbose = true;
  fit_camera_to_xyz_ht(opt.parse_ecef, parsed_cam_ctr, input_cam_ctr,
                       opt.camera_type, opt.refine_camera,  
                       xyz_vec, opt.pixel_values, 
                       opt.cam_height, opt.cam_weight, opt.cam_ctr_weight, datum,
                       verbose, out_cam);
    
  return;
}

// Read a pinhole camera from Planet's json file format (*_pinhole.json). Then
// the WGS84 datum is assumed.
void read_pinhole_from_json(Options const& opt, vw::cartography::Datum & datum,
                            boost::shared_ptr<CameraModel> & out_cam) {

  datum.set_well_known_datum("WGS84");
  
  std::ifstream f(opt.input_camera);
  json j = json::parse(f);

  // Parse the focal length and optical center. Negate the focal
  // length to make it positive. We adjust for that later.
  json const& cam = j["P_camera"];
  double fx = -cam[0][0].get<double>();
  double fy = -cam[1][1].get<double>();
  double ox = cam[0][2].get<double>();
  double oy = cam[1][2].get<double>();

  json const& exterior = j["exterior_orientation"];
  double ecef_x = exterior["x_ecef_meters"].get<double>();
  double ecef_y = exterior["y_ecef_meters"].get<double>();
  double ecef_z = exterior["z_ecef_meters"].get<double>();

  // Following the Planet convention of naming things
  vw::Matrix<double> extrinsic = json_mat(j["P_extrinsic"], 4, 4);
  vw::Matrix<double> intrinsic = json_mat(j["P_intrinsic"], 4, 4);

  // Adjust for the fact that Planet likes negative focal lengths, while
  // vw::camera::PinholeModel uses positive values.
  vw::Matrix<double> flip;
  flip.set_identity(4);
  flip(0, 0) = -1;
  flip(1, 1) = -1;
      
  // Create a blank pinhole model and get an alias to it
  out_cam.reset(new vw::camera::PinholeModel());
  PinholeModel & pin = *((PinholeModel*)out_cam.get());

  // Populate the model
  pin.set_pixel_pitch(1.0); // not necessary, but better be explicit
  pin.set_focal_length(vw::Vector2(fx, fy));
  pin.set_point_offset(vw::Vector2(ox, oy));

  pin.set_camera_center(vw::Vector3(ecef_x, ecef_y, ecef_z));

  vw::Matrix<double> world2cam = flip * intrinsic * extrinsic;
  vw::Matrix<double> cam2world = inverse(world2cam);
  pin.set_camera_pose(submatrix(cam2world, 0, 0, 3, 3));
}

int main(int argc, char * argv[]){
  
  Options opt;
  try {
    
    handle_arguments(argc, argv, opt);
    
    boost::shared_ptr<CameraModel> out_cam;
    vw::cartography::Datum datum;

    // Some of the numbers we print need high precision
    vw_out().precision(17);
    
    if (!opt.input_pinhole) {
      // Create a pinhole camera using user-specified options.
      form_pinhole_camera(opt, datum, out_cam);
    } else {
      // Read a pinhole camera from Planet's json file format (*_pinhole.json). Then
      // the WGS84 datum is assumed. Ignore all other input options.
      read_pinhole_from_json(opt, datum, out_cam);
    }
    
    vw::Vector3 llh = datum.cartesian_to_geodetic(out_cam->camera_center(Vector2()));
    vw_out() << "Output camera center lon, lat, and height above datum: " << llh << std::endl;
    vw_out() << "Writing: " << opt.camera_file << std::endl;
    if (opt.camera_type == "opticalbar")
      ((vw::camera::OpticalBarModel*)out_cam.get())->write(opt.camera_file);
    else 
      ((vw::camera::PinholeModel*)out_cam.get())->write(opt.camera_file);
    
  } ASP_STANDARD_CATCHES;
  
  return 0;
}
