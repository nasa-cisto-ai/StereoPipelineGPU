#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file dem_mosaic.cc
///

// A tool to mosaic and blend DEMs, and output the mosaic as tiles.

// Note 1: In practice, the tool may be more efficient if the entire
// mosaic is written out as one single large image, rather than being
// broken up into tiles. To achieve that, just specify to the tool a
// very large tile size, and use 0 for the tile index in the command
// line options.

// Note 2: The tool can be high on memory usage, so processes for
// individual tiles may need to be run on separate machines.

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>

#include <vw/FileIO/DiskImageManager.h>
#include <vw/Image/InpaintView.h>
#include <vw/Image/Algorithms2.h>
#include <vw/Image/Filter.h>
#include <vw/Cartography/GeoTransform.h>

#include <boost/math/special_functions/fpclassify.hpp>
#include <boost/math/special_functions/erf.hpp>
#include <boost/program_options.hpp>
#include <boost/filesystem/convenience.hpp>

#include <iostream>
#include <fstream>
#include <iomanip>
#include <string>
#include <vector>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <time.h>
#include <limits>
#include <algorithm>

using namespace vw; // TODO(oalexan1): Remove this namespace
using namespace vw::cartography;
namespace po = boost::program_options;
namespace fs = boost::filesystem;

// This tool casts all input DEMs to float. The processing is done in double
// precision though. 
typedef float RealT;

// This is used for various tolerances
double g_tol = 1e-6;

// TODO: Fold modifications into VW!
template<class ImageT>
void centerline_weights2(ImageT const& img, ImageView<double> & weights,
                         double hole_fill_value=0, double border_fill_value=-1, 
                         BBox2i roi=BBox2i()){

  int numRows = img.rows();
  int numCols = img.cols();

  // Arrays to be returned out of this function
  std::vector<double> hCenterLine  (numRows, 0);
  std::vector<double> hMaxDistArray(numRows, 0);
  std::vector<double> vCenterLine  (numCols, 0);
  std::vector<double> vMaxDistArray(numCols, 0);

  std::vector<int> minValInRow(numRows, 0);
  std::vector<int> maxValInRow(numRows, 0);
  std::vector<int> minValInCol(numCols, 0);
  std::vector<int> maxValInCol(numCols, 0);

  for (int k = 0; k < numRows; k++){
    minValInRow[k] = numCols;
    maxValInRow[k] = 0;
  }
  for (int col = 0; col < numCols; col++){
    minValInCol[col] = numRows;
    maxValInCol[col] = 0;
  }

  // Note that we do just a single pass through the image to compute
  // both the horizontal and vertical min/max values.
  for (int row = 0 ; row < numRows; row++) {
    for (int col = 0; col < numCols; col++) {

      if (!is_valid(img(col,row))) continue;
      
      // Record the first and last valid column in each row
      if (col < minValInRow[row]) minValInRow[row] = col;
      if (col > maxValInRow[row]) maxValInRow[row] = col;
      
      // Record the first and last valid row in each column
      if (row < minValInCol[col]) minValInCol[col] = row;
      if (row > maxValInCol[col]) maxValInCol[col] = row;   
    }
  }
  
  // For each row, record central column and the column width
  for (int row = 0; row < numRows; row++) {
    hCenterLine   [row] = (minValInRow[row] + maxValInRow[row])/2.0;
    hMaxDistArray [row] =  maxValInRow[row] - minValInRow[row];
    if (hMaxDistArray[row] < 0){
      hMaxDistArray[row]=0;
    }
  }

  // For each row, record central column and the column width
  for (int col = 0 ; col < numCols; col++) {
    vCenterLine   [col] = (minValInCol[col] + maxValInCol[col])/2.0;
    vMaxDistArray [col] =  maxValInCol[col] - minValInCol[col];
    if (vMaxDistArray[col] < 0){
      vMaxDistArray[col]=0;
    }
  }

  BBox2i output_bbox = roi;
  if (roi.empty())
    output_bbox = bounding_box(img);

  // Compute the weighting for each pixel in the image
  weights.set_size(output_bbox.width(), output_bbox.height());
  fill(weights, 0);
  
  for (int row = output_bbox.min().y(); row < output_bbox.max().y(); row++){
    for (int col = output_bbox.min().x(); col < output_bbox.max().x(); col++){
      bool inner_row = ((row >= minValInCol[col]) && (row <= maxValInCol[col]));
      bool inner_col = ((col >= minValInRow[row]) && (col <= maxValInRow[row]));
      bool inner_pixel = inner_row && inner_col;
      vw::Vector2 pix(col, row);
      double new_weight = 0; // Invalid pixels usually get zero weight
      if (is_valid(img(col,row))) {
        double weight_h = compute_line_weights(pix, true,  hCenterLine, hMaxDistArray);
        double weight_v = compute_line_weights(pix, false, vCenterLine, vMaxDistArray);
        new_weight = weight_h*weight_v;
      }
      else { // Invalid pixel
        if (inner_pixel)
          new_weight = hole_fill_value;
        else // Border pixel
          new_weight = border_fill_value;
      }
      weights(col-output_bbox.min().x(), row-output_bbox.min().y()) = new_weight;
      
    }
  }

} // End function centerline_weights2

// An S-shaped function. Value at 0 is 0. Value at M is M.
// Flat before 0 and after M. Higher value of L means
// more flatness at the ends, but higher growth
// in the middle.
double S_shape(double x, double M, double L){
  if (x <= 0) return 0;
  if (x >= M) return M;
  return 0.5*M*(1 + boost::math::erf (0.5*sqrt(M_PI) * (2*x*L/M - L)));
}

// Function for highlighting spots of data
template<class PixelT>
class NotNoDataFunctor {
  typedef typename CompoundChannelType<PixelT>::type channel_type;
  channel_type m_nodata;
  typedef ChannelRange<channel_type> range_type;
public:
  NotNoDataFunctor(channel_type nodata) : m_nodata(nodata) {}

  template <class Args> struct result {
    typedef channel_type type;
  };

  inline channel_type operator()(channel_type const& val) const {
    return (val != m_nodata && !std::isnan(val))? range_type::max() : range_type::min();
  }
};

template <class ImageT, class NoDataT>
UnaryPerPixelView<ImageT,UnaryCompoundFunctor<NotNoDataFunctor<typename ImageT::pixel_type>, typename ImageT::pixel_type>  >
inline notnodata(ImageViewBase<ImageT> const& image, NoDataT nodata) {
  typedef UnaryCompoundFunctor<NotNoDataFunctor<typename ImageT::pixel_type>, typename ImageT::pixel_type> func_type;
  func_type func(nodata);
  return UnaryPerPixelView<ImageT,func_type>(image.impl(), func);
}

// Set nodata pixels to 0 and valid data pixels to something big.
template<class PixelT>
struct BigOrZero: public ReturnFixedType<PixelT> {
  PixelT m_nodata;
  BigOrZero(PixelT nodata):m_nodata(nodata){}
  double operator() (PixelT const& pix) const {
    if (pix != m_nodata && !std::isnan(pix)) return 1e+8;
    return 0;
  }
};

void blur_weights(ImageView<double> & weights, double sigma){

  if (sigma <= 0)
    return;

  // Blur the weights. To try to make the weights not drop much at the
  // boundary, expand the weights with zero, blur, crop back to the
  // original region.

  // It is highly important to note that blurring can increase the weights
  // at the boundary, even with the extension done above. Erosion before
  // blurring does not help with that, as for weights with complicated
  // boundary erosion can wipe things in a non-uniform way leaving
  // huge holes. To get smooth weights, if really desired one should
  // use the weights-exponent option.

  int half_kernel = vw::compute_kernel_size(sigma)/2;
  int extra = half_kernel + 1; // to guarantee we stay zero at boundary

  int cols = weights.cols(), rows = weights.rows();

  ImageView<double> extra_wts(cols + 2*extra, rows + 2*extra);
  fill(extra_wts, 0);
  for (int col = 0; col < cols; col++) {
    for (int row = 0; row < rows; row++) {
      if (weights(col,row) > 0)
        extra_wts(col + extra, row + extra) = weights(col, row);
      else
        extra_wts(col + extra, row + extra) = 0;
    }
  }

  ImageView<double> blurred_wts = gaussian_filter(extra_wts, sigma);

  // Copy back.  The weights must not grow. In particular, where the
  // original weights were zero, the new weights must also be zero, as
  // at those points there is no DEM data.
  for (int col = 0; col < cols; col++) {
    for (int row = 0; row < rows; row++) {
      if (weights(col, row) > 0) {
        weights(col, row) = blurred_wts(col + extra, row + extra);
      }
      //weights(col, row) = std::min(weights(col, row), blurred_wts(col + extra, row + extra));
    }
  }

}

BBox2 custom_point_to_pixel_bbox(GeoReference const& georef, BBox2 const& ptbox){

  // Given the corners in the projected space, find the pixel corners.
  // If the biggest pixel value is say 42, the image must have 43
  // pixels.  And if the biggest pixel value is 42.1 or 42.9, the
  // image must have 44 pixels.  That's why below we use
  // ceil(pix_box.max() + Vector2(1, 1)).

  // This differs a bit from the point_to_pixel_bbox() function in
  // GeoReferenceBase.cc.

  // TODO: Maybe that function can be made to imitate this one. 

  vw::BBox2 pix_box;
  vw::Vector2 cr[] = {ptbox.min(), ptbox.max(),
		  vw::Vector2(ptbox.min().x(), ptbox.max().y()),
		  vw::Vector2(ptbox.max().x(), ptbox.min().y())};
  for (int icr = 0; icr < (int)(sizeof(cr)/sizeof(Vector2)); icr++)
    pix_box.grow(georef.point_to_pixel(cr[icr]));
  
  // If the corner is actually very close to an integer number, we
  // assume it should in fact be integer but got moved a bit due to
  // numerical error. Then we set it to integer. This ensures that
  // when we mosaic a single DEM we get its corners to be the same as
  // the originals rather than moved by a slight offset.
  if (norm_2(pix_box.max() - round(pix_box.max())) < g_tol) 
    pix_box.max() = round(pix_box.max());

  pix_box.max() = ceil(pix_box.max() + vw::Vector2(1, 1));
  
  return pix_box;
}

GeoReference read_georef(std::string const& file){
  // Read a georef, and check for success
  GeoReference geo;
  bool is_good = read_georeference(geo, file);
  if (!is_good)
    vw_throw(ArgumentErr() << "No georeference found in " << file << ".\n");
  return geo;
}

std::string processed_proj4(std::string const& srs){
  // Apparently functionally identical proj4 strings can differ in
  // subtle ways, such as an extra space, etc. For that reason, must
  // parse and process any srs string before comparing it with another string.
  GeoReference georef;
  bool  have_user_datum = false, have_input_georef = false;
  Datum user_datum;
  asp::set_srs_string(srs, have_user_datum, user_datum,
                      have_input_georef, georef);
  return georef.overall_proj4_str();
}

struct Options: vw::GdalWriteOptions {
  std::string dem_list_file, out_prefix, target_srs_string,
    output_type, tile_list_str, this_dem_as_reference;
  std::vector<std::string> dem_files;
  double tr, geo_tile_size;
  bool   has_out_nodata, force_projwin;
  double out_nodata_value;
  int    tile_size, tile_index, erode_len, priority_blending_len,
         extra_crop_len, hole_fill_len, block_size, save_dem_weight,
         fill_num_passes;
  double weights_exp, weights_blur_sigma, dem_blur_sigma;
  double nodata_threshold, fill_search_radius, fill_power, fill_percent;
  bool   first, last, min, max, block_max, mean, stddev, median, nmad,
    count, tap, save_index_map, use_centerline_weights,
         first_dem_as_reference, propagate_nodata, no_border_blend;
  std::set<int> tile_list;
  BBox2 projwin;
  Options(): tr(0), geo_tile_size(0), has_out_nodata(false), force_projwin(false), 
             tile_index(-1), erode_len(0), priority_blending_len(0), extra_crop_len(0),
             hole_fill_len(0), block_size(0), save_dem_weight(-1), 
             fill_search_radius(0), fill_power(0), fill_percent(0), fill_num_passes(0),
             weights_exp(0), weights_blur_sigma(0.0), dem_blur_sigma(0.0),
             nodata_threshold(std::numeric_limits<double>::quiet_NaN()),
             first(false), last(false), min(false), max(false), block_max(false),
             mean(false), stddev(false), median(false), nmad(false),
             count(false), save_index_map(false), tap(false),
             use_centerline_weights(false), first_dem_as_reference(false), projwin(BBox2()) {}
};

/// Return the number of no-blending options selected.
int no_blend(Options const& opt){
  return int(opt.first) + int(opt.last) + int(opt.min) + int(opt.max)
    + int(opt.mean) + int(opt.stddev) + int(opt.median)
    + int(opt.nmad) + int(opt.count) + int(opt.block_max);
}

std::string tile_suffix(Options const& opt){
  std::string ans;
  if (opt.first) ans     = "-first";
  if (opt.last) ans      = "-last";
  if (opt.min) ans       = "-min";
  if (opt.max) ans       = "-max";
  if (opt.block_max) ans = "-block-max";
  if (opt.mean) ans      = "-mean";
  if (opt.stddev) ans    = "-stddev";
  if (opt.median) ans    = "-median";
  if (opt.nmad) ans      = "-nmad";
  if (opt.count) ans     = "-count";
  if (opt.save_index_map)       ans += "-index-map";
  if (opt.save_dem_weight >= 0) ans += "-weight-dem-index-" + stringify(opt.save_dem_weight);

  return ans;
}

/// Class that does the actual image processing work
class DemMosaicView: public ImageViewBase<DemMosaicView>{
  int m_cols, m_rows, m_bias;
  Options                   const& m_opt;              // alias
  DiskImageManager<RealT>        & m_imgMgr;           // alias
  std::vector<GeoReference> const& m_georefs;          // alias
  GeoReference                     m_out_georef;
  std::vector<double>       const& m_nodata_values;    // alias
  std::vector<vw::BBox2i>          const& m_dem_pixel_bboxes; // alias
  long long int                  & m_num_valid_pixels; // alias, to populate on output
  vw::Mutex                      & m_count_mutex;      // alias, a lock for m_num_valid_pixels

public:
  DemMosaicView(int cols, int rows, int bias,
                Options                   const& opt,
                DiskImageManager<RealT>        & imgMgr,
                std::vector<GeoReference> const& georefs,
                GeoReference              const& out_georef,
                std::vector<double>       const& nodata_values,
                std::vector<BBox2i>       const& dem_pixel_bboxes,
                long long int                  & num_valid_pixels,
                vw::Mutex                      & count_mutex):
    m_cols(cols), m_rows(rows), m_bias(bias), m_opt(opt),
    m_imgMgr(imgMgr), m_georefs(georefs),
    m_out_georef(out_georef), m_nodata_values(nodata_values),
    m_dem_pixel_bboxes(dem_pixel_bboxes), m_num_valid_pixels(num_valid_pixels),
    m_count_mutex(count_mutex) {

    // How many valid pixels we will have
    m_num_valid_pixels = 0;
    
    if (imgMgr.size() != georefs.size()       ||
        imgMgr.size() != nodata_values.size() ||
        imgMgr.size() != dem_pixel_bboxes.size())
      vw_throw(ArgumentErr() << "Inputs expected to have the same size do not.\n");

    // Sanity check, see if datums differ, then the tool won't work
    const double out_major_axis = m_out_georef.datum().semi_major_axis();
    const double out_minor_axis = m_out_georef.datum().semi_minor_axis();
    for (int i = 0; i < (int)m_georefs.size(); i++) {
      double this_major_axis = m_georefs[i].datum().semi_major_axis();
      double this_minor_axis = m_georefs[i].datum().semi_minor_axis();
      if (std::abs(this_major_axis - out_major_axis) > 0.1 || 
          std::abs(this_minor_axis - out_minor_axis) > 0.1 ||
          m_georefs[i].datum().meridian_offset()
          != m_out_georef.datum().meridian_offset()){
        vw_throw(NoImplErr() << "Mosaicking of DEMs with differing datum radii "
                 << " or meridian offsets is not implemented. Datums encountered:\n"
                 << m_georefs[i].datum() << "\n"
                 <<  m_out_georef.datum() << "\n");
      }
      if (m_georefs[i].datum().name() != m_out_georef.datum().name() &&
          this_major_axis == out_major_axis &&
          this_minor_axis == out_minor_axis &&
          m_georefs[i].datum().meridian_offset() == m_out_georef.datum().meridian_offset()){
        vw_out(WarningMessage) << "Found DEMs with the same radii and meridian offsets, "
                               << "but different names: "
                               << m_georefs[i].datum().name() << " and "
                               << m_out_georef.datum().name() << "\n";
      }
    }
  }

  // Boilerplate
  typedef RealT      pixel_type;
  typedef pixel_type result_type;
  typedef ProceduralPixelAccessor<DemMosaicView> pixel_accessor;
  inline int cols  () const { return m_cols; }
  inline int rows  () const { return m_rows; }
  inline int planes() const { return 1; }
  inline pixel_accessor origin() const { return pixel_accessor(*this, 0, 0); }

  inline pixel_type operator()(double/*i*/, double/*j*/, int/*p*/ = 0) const {
    vw_throw(NoImplErr() << "DemMosaicView::operator()(...) is not implemented");
    return pixel_type();
  }

  typedef CropView<ImageView<pixel_type> > prerasterize_type;
  inline prerasterize_type prerasterize(BBox2i bbox) const {

    BBox2i orig_box = bbox;

    // Get a shorthand for this
    const bool use_priority_blend = (m_opt.priority_blending_len > 0);
    
    // When doing priority blending, we will do all the work in the
    // output pixels domain. Hence we need to take into account the
    // bias here rather than later.
    if (use_priority_blend)
      bbox.expand(m_bias + BilinearInterpolation::pixel_buffer + 1);

    // We will do all computations in double precision, regardless
    // of the precision of the inputs, for increased accuracy.
    // - The image data buffers are initialized here
    typedef PixelGrayA<double> DoubleGrayA;
    ImageView<double> tile   (bbox.width(), bbox.height()); // the output tile (in most cases)
    ImageView<double> weights(bbox.width(), bbox.height()); // accumulated weights (in most cases)
    fill(tile, m_opt.out_nodata_value);
    fill(weights, 0.0);

    // True if we won't be doing any DEM blending.
    bool noblend = (no_blend(m_opt) > 0);

    // A vector of images the size of the output tile.
    // - Used for median, nmad, and stddev calculation.
    std::vector<ImageView<double>> tile_vec, weight_vec;
    std::vector<std::string> dem_vec;
    if (m_opt.median || m_opt.nmad) // Store each input separately
      tile_vec.reserve(m_imgMgr.size());
    if (m_opt.stddev) { // Need one working image
      tile_vec.push_back(ImageView<double>(bbox.width(), bbox.height()));
      // Each pixel starts at zero, nodata is handled later
      fill(tile_vec[0], 0.0);
      fill(tile,        0.0);
    }
    if (use_priority_blend) { // Store each weight separately
      tile_vec.reserve  (m_imgMgr.size());
      weight_vec.reserve(m_imgMgr.size());
    }

    // This will ensure that pixels from earlier images are
    // mostly used unmodified except being blended at the boundary.
    vw::ImageView<double> weight_modifier;
    if (use_priority_blend) {
      weight_modifier = vw::ImageView<double>(bbox.width(), bbox.height());
      fill(weight_modifier, std::numeric_limits<double>::max());
    }

    // For saving the weights
    std::vector<int> clip2dem_index;
    ImageView<double> saved_weight;
    if (m_opt.save_dem_weight >= 0) {
      saved_weight = ImageView<double>(bbox.width(), bbox.height());
      fill(saved_weight, 0.0);
    }

    // For saving the index map
    ImageView<double> index_map;
    if (m_opt.save_index_map) {
      index_map = ImageView<double>(bbox.width(), bbox.height());
      fill(index_map, m_opt.out_nodata_value);

      // Sanity check: the output no-data value must not equal to
      // any of the indices in the map, as then the two cannot be
      // distinguished.
      for (int dem_iter = 0; dem_iter < (int)m_imgMgr.size(); dem_iter++){
        if (dem_iter == m_opt.out_nodata_value) 
          vw_throw(ArgumentErr() << "Cannot have the output no-data value equal to "
                   << m_opt.out_nodata_value
                   << " as this is one of the indices being saved in the index map.\n");
      }
    }

    ImageView<double> first_dem;
    ImageView<double> local_wts_orig;

    // Loop through all input DEMs
    for (int dem_iter = 0; dem_iter < (int)m_imgMgr.size(); dem_iter++) {

      // Load the information for this DEM
      GeoReference georef        = m_georefs         [dem_iter];
      BBox2i       dem_pixel_box = m_dem_pixel_bboxes[dem_iter];

      // The GeoTransform will hide the messy details of conversions
      // from pixels to points and lon-lat.
      GeoTransform geotrans(georef, m_out_georef, dem_pixel_box, bbox);

      // Get the tile bbox in the frame of the current input DEM
      BBox2 in_box = geotrans.reverse_bbox(bbox);

      // Grow to account for blending and erosion length, etc.  If
      // priority blending length was positive, we've already expanded 'bbox'.
      if (!use_priority_blend)
        in_box.expand(m_bias + BilinearInterpolation::pixel_buffer + 1);

      in_box.crop(dem_pixel_box);
      if (in_box.width() == 1 || in_box.height() == 1){
        // Grassfire likes to have width of at least 2
        in_box.expand(1);
        in_box.crop(dem_pixel_box);
      }
      if (in_box.width() <= 1 || in_box.height() <= 1)
        continue; // No overlap with this tile, skip to the next DEM.

      if (m_opt.median || m_opt.nmad || use_priority_blend || m_opt.block_max){
        // Must use a blank tile each time
        fill(tile, m_opt.out_nodata_value);
        fill(weights, 0.0);
      }

      // Crop the disk dem to a 2-channel in-memory image. First
      // channel is the image pixels, second will be the weights.
      ImageViewRef<double> disk_dem = pixel_cast<double>(m_imgMgr.get_handle(dem_iter, bbox));
      ImageView<DoubleGrayA> dem    = crop(disk_dem, in_box);

      if (m_opt.first_dem_as_reference && dem_iter == 0) {
        // We need to keep the first DEM, to use it as ref
        // when merging in the blended DEM
        first_dem = crop(disk_dem, bbox);
      }

      std::string dem_name = m_imgMgr.get_file_name(dem_iter);

      // If the nodata_threshold is specified, all values no more than this
      // will be invalidated.
      double nodata_value = m_nodata_values[dem_iter];
      if (!boost::math::isnan(m_opt.nodata_threshold)) {
        nodata_value = m_opt.nodata_threshold;
        for (int col = 0; col < dem.cols(); col++) {
          for (int row = 0; row < dem.rows(); row++) {
            if (dem(col, row)[0] <= nodata_value) {
              dem(col, row)[0] = nodata_value;
            }
          }
        }
      }

      if (m_opt.first_dem_as_reference && dem_iter == 0) {
        //TODO: Should be a function!
        // Convert to the output nodata value
        for (int col = 0; col < first_dem.cols(); col++) {
          for (int row = 0; row < first_dem.rows(); row++) {
            if (first_dem(col, row) == nodata_value) {
              first_dem(col, row) = m_opt.out_nodata_value;
            }
          }
        }
      }

      // Fill holes. This happens here, in the expanded tile, to
      // ensure we catch holes which are partially outside the tile
      // being processed.
      if (m_opt.hole_fill_len > 0)
        dem = apply_mask(vw::fill_holes_grass
                         (create_mask(select_channel(dem, 0), nodata_value),
                          m_opt.hole_fill_len),
                         nodata_value);

      // Fill nodata based on radius. There is a sanity check that ensures we don't
      // do both this and the hole filling above.
      if (m_opt.fill_search_radius > 0.0) {
        dem = apply_mask(fillNodataWithSearchRadius
        (create_mask(select_channel(dem, 0), nodata_value),
          m_opt.fill_search_radius, m_opt.fill_power, m_opt.fill_percent, 
          m_opt.fill_num_passes),
          nodata_value);
      }

      // Fill-in no-data values a bit and blur. If just the blurring is used,
      // it will choke on no-data values, leaving large holes around each,
      // hence the need to fill a little.
      if (m_opt.dem_blur_sigma > 0.0) {
        int kernel_size = vw::compute_kernel_size(m_opt.dem_blur_sigma);
        dem = apply_mask(gaussian_filter(fill_nodata_with_avg
                                         (create_mask(select_channel(dem, 0), nodata_value),
                                          kernel_size),
                                         m_opt.dem_blur_sigma),
                         nodata_value);
      }
      
      // Mark the handle to the image as not in use, though we still
      // keep that image file open, for increased performance, unless
      // their number becomes too large.
      m_imgMgr.release(dem_iter);

      if (dem_iter == 0 && m_opt.this_dem_as_reference != "") {
        // We won't actually use this DEM, we just do all in reference to it.
        continue;
      }

      // Compute linear weights
      ImageView<double> local_wts = grassfire(notnodata(select_channel(dem, 0), nodata_value),
                                              m_opt.no_border_blend);
      local_wts_orig = local_wts;
      if (m_opt.use_centerline_weights) {
        // Erode based on grassfire weights, and then overwrite the grassfire
        // weights with centerline weights
        ImageView<DoubleGrayA> dem2 = copy(dem);
        for (int col = 0; col < dem2.cols(); col++) {
          for (int row = 0; row < dem2.rows(); row++) {
            if (local_wts(col, row) <= m_opt.erode_len) {
              dem2(col, row) = DoubleGrayA(nodata_value);
            }
          }
        }
        // TODO: Generalize this modification and move it to VW!!!
        centerline_weights2
                (create_mask_less_or_equal(select_channel(dem2, 0), nodata_value),
                 local_wts, -1.0);
      } // End centerline weights case

      // If we don't limit the weights from above, we will have tiling artifacts,
      // as in different tiles the weights grow to different heights since
      // they are cropped to different regions. For priority blending length,
      // we'll do this process later, as the bbox is obtained differently in that case.
      if (!use_priority_blend) {
        for (int col = 0; col < local_wts.cols(); col++) {
          for (int row = 0; row < local_wts.rows(); row++) {
            local_wts(col, row) = std::min(local_wts(col, row), double(m_bias));
          }
        }
      }

      // Erode. We already did that if centerline weights are used.
      if (!m_opt.use_centerline_weights){
        int max_cutoff = max_pixel_value(local_wts);
        int min_cutoff = m_opt.erode_len;
        if (max_cutoff <= min_cutoff)
          max_cutoff = min_cutoff + 1; // precaution
        local_wts = clamp(local_wts - min_cutoff, 0.0, max_cutoff - min_cutoff);
      }
      
      // Blur the weights. If priority blending length is on, we'll do the blur later,
      // after weights from different DEMs are combined.
      if (m_opt.weights_blur_sigma > 0 && !use_priority_blend)
        blur_weights(local_wts, m_opt.weights_blur_sigma);

      // Raise to the power. Note that when priority blending length is positive, we
      // delay this process.
      if (m_opt.weights_exp != 1 && !use_priority_blend) {
        for (int col = 0; col < dem.cols(); col++){
          for (int row = 0; row < dem.rows(); row++){
            if (local_wts(col, row) > 0)
              local_wts(col, row) = pow(local_wts(col, row), m_opt.weights_exp);
          }
        }
      }

#if 0
      // Dump the weights
      std::ostringstream os;
      os << "weights_" << dem_iter << ".tif";
      vw_out() << "Writing: " << os.str() << std::endl;
      bool has_georef = true, has_nodata = true;
      block_write_gdal_image(os.str(), local_wts,
			     has_georef, georef,
			     has_nodata, -100,
			     vw::GdalWriteOptions(),
			     TerminalProgressCallback("asp", ""));
#endif

      // TODO: Function call!
      // Set the weights in the alpha channel
      for (int col = 0; col < dem.cols(); col++){
        for (int row = 0; row < dem.rows(); row++){
          dem(col, row).a() = local_wts(col, row);
        }
      }

      // Prepare the DEM for interpolation
      ImageViewRef<DoubleGrayA> interp_dem
        = interpolate(dem, BilinearInterpolation(), ConstantEdgeExtension());

      // Loop through each output pixel
      for (int c = 0; c < bbox.width(); c++){
        for (int r = 0; r < bbox.height(); r++){

          // Coordinates in the output mosaic
          vw::Vector2 out_pix(c +  bbox.min().x(), r +  bbox.min().y());
          // Coordinate in this input DEM
          vw::Vector2 in_pix = geotrans.reverse(out_pix);

          // Input DEM pixel relative to loaded bbox
          double x = in_pix[0] - in_box.min().x();
          double y = in_pix[1] - in_box.min().y();
          DoubleGrayA pval;

          int i0 = round(x),  // Round to nearest integer location
              j0 = round(y);
          if ((fabs(x-i0) < g_tol) && (fabs(y-j0) < g_tol) &&
              ((i0 >= 0) && (i0 <= dem.cols()-1) &&
               (j0 >= 0) && (j0 <= dem.rows()-1))){

            // A lot of care is needed here. We are at an integer
            // pixel, save for numerical error. Just borrow pixel's
            // value, and don't interpolate. Interpolation can result
            // in invalid pixels if the current pixel is valid but its
            // neighbors are not. It can also make it appear is if the
            // current indices are out of bounds while in fact they
            // are barely so.
            pval = dem(i0, j0);

          }else{ // We are not right on an integer pixel and we need to interpolate

            // Below must use x <= cols()-1 as x is double
            bool is_good = ((x >= 0) && (x <= dem.cols()-1) && // TODO: should be an image function!
		            (y >= 0) && (y <= dem.rows()-1));
            if (!is_good)
              continue; // Outside the loaded DEM bounds, skip to the next pixel

            // If we have weights of 0, that means there are invalid pixels, so skip this point.
            int i0 = (int)floor(x), j0 = (int)floor(y);
            int i1 = (int)ceil(x),  j1 = (int)ceil(y);
            bool nodata = ((dem(i0, j0).a() == 0) || (dem(i1, j0).a() == 0) ||
                           (dem(i0, j1).a() == 0) || (dem(i1, j1).a() == 0));
            bool border = ((dem(i0, j0).a() <  0) || (dem(i1, j0).a() <  0) ||
                           (dem(i0, j1).a() <  0) || (dem(i1, j1).a() <  0));

            if (nodata || border) {
              pval.v() = 0;
              pval.a() = -1; // Flag as border

              if (m_opt.propagate_nodata && !border)
                pval.a() = 0; // Flag as nodata

            } else
              pval = interp_dem(x, y); // Things checked out, do the interpolation.
          }
          // Seperate the value and alpha for this pixel.
          double val = pval.v();
          double wt  = pval.a();

          if (use_priority_blend) {
            // The priority blending, pixels from earlier DEMs at this location
            // are used unmodified unless close to that DEM boundary.
            wt = std::min(weight_modifier(c, r), wt);

            // Now ensure that the current DEM values will be used
            // unmodified unless close to the boundary for subsequent
            // DEMs. The weight w2 will be 0 well inside the DEM, and
            // increase towards the boundary.
            double wt2 = wt;
            wt2 = std::max(0.0, m_opt.priority_blending_len - wt2);
            weight_modifier(c, r) = std::min(weight_modifier(c, r), wt2);
          }

          // If point is in-bounds and nodata, make sure this point stays 
          //  at nodata even if other DEMS contain it.
          if ((wt == 0) && m_opt.propagate_nodata) {
            tile   (c, r) = 0;
            weights(c, r) = -1.0;
          }

          if (wt <= 0.0)
            continue; // No need to continue if the weight is zero

          // Check if the current output value at this pixel is nodata
          bool is_nodata = ((tile(c, r) == m_opt.out_nodata_value));

          // Initialize the tile if not done already.
          // Init to zero not needed with some types.
          if (!m_opt.stddev && !m_opt.median && !m_opt.nmad && !m_opt.min && !m_opt.max &&
              !use_priority_blend){
            if (is_nodata){
              tile   (c, r) = 0;
              weights(c, r) = 0.0;
            }
          }

          // Update the output value according to the commanded mode
          if ((m_opt.first && is_nodata)                      ||
               m_opt.last                                     ||
               (m_opt.min && (val < tile(c, r) || is_nodata)) ||
               (m_opt.max && (val > tile(c, r) || is_nodata)) ||
               m_opt.median || m_opt.nmad || 
               use_priority_blend   || m_opt.block_max){
            // --> Conditions where we replace the current value
            tile   (c, r) = val;
            weights(c, r) = wt;

            // In these cases, the saved weight will be 1 or 0, since either
            // a given DEM gives it all, or nothing at all.
            if (m_opt.save_dem_weight >= 0 && (m_opt.first || m_opt.last ||
                                               m_opt.min   || m_opt.max))
              saved_weight(c, r) = (m_opt.save_dem_weight == dem_iter);

            // In these cases, the saved weight will be 1 or 0, since either
            // a given DEM gives it all, or nothing at all.
            if (m_opt.save_index_map && (m_opt.first || m_opt.last ||
                                         m_opt.min   || m_opt.max))
              index_map(c, r) = dem_iter;

          }else if (m_opt.mean){ // Mean --> Accumulate the value
            tile(c, r) += val;
            weights(c, r)++;

            if (m_opt.save_dem_weight == dem_iter)
              saved_weight(c, r) = 1;

          }else if (m_opt.count){ // Count --> Increment the value
            tile(c, r)++;
            weights(c, r) += wt;
          }else if (m_opt.stddev){ // Standard Deviation --> Keep running calculation
            weights(c, r) += 1.0;
            double curr_mean = tile_vec[0](c,r);
            double delta     = val - curr_mean;
            curr_mean     += delta / weights(c, r);
            double newVal = tile(c, r) + delta*(val - curr_mean);
            tile(c, r)    = newVal;
            tile_vec[0](c,r) = curr_mean;
          }else if (!noblend){ // Blending --> Weighted average
            tile(c, r) += wt*val;
            weights(c, r) += wt;
            if (m_opt.save_dem_weight == dem_iter)
              saved_weight(c, r) = wt;
          }

        } // End col loop
      } // End row loop

      // For the median option, keep a copy of the output tile for each input DEM!
      // Also do it for max per block.
      // - This will be memory intensive. 
      if (m_opt.median || m_opt.nmad || m_opt.block_max) {
        tile_vec.push_back(copy(tile));
        dem_vec.push_back(dem_name);
      }
      
      // For priority blending, need also to keep all tiles, but also the weights
      if (use_priority_blend){
        tile_vec.push_back(copy(tile));
        weight_vec.push_back(copy(weights));
      }
      
      if (use_priority_blend || m_opt.save_index_map)
	clip2dem_index.push_back(dem_iter);
      
    } // End iterating over DEMs

    // Divide by the weights in blend, mean
    if (!noblend || m_opt.mean){
      for (int c = 0; c < bbox.width(); c++){ // Iterate over all pixels!
        for (int r = 0; r < bbox.height(); r++){
          if (weights(c, r) > 0)
            tile(c, r) /= weights(c, r);

          //if (m_opt.save_dem_weight >= 0 && weights(c, r) > 0)
          //  saved_weight(c, r) /= weights(c, r);

        } // End row loop
      } // End col loop
    } // End dividing case


    // Finish stddev calculations
    if (m_opt.stddev){
      for (int c = 0; c < bbox.width(); c++){ // Iterate over all pixels!
        for (int r = 0; r < bbox.height(); r++){

          if (weights(c, r) > 1.0){
            tile(c, r) = sqrt(tile(c, r) / (weights(c, r) - 1.0));
          } else { // Invalid pixel!
            tile(c, r) = m_opt.out_nodata_value;
          }
        } // End row loop
      } // End col loop
    } // End stddev case

    // For the median and nmad operations
    if (m_opt.median || m_opt.nmad){
      // Init output pixels to nodata
      fill(tile, m_opt.out_nodata_value);
      std::vector<double> vals, vals_all(tile_vec.size());
      // Iterate through all pixels
      for (int c = 0; c < bbox.width(); c++){
        for (int r = 0; r < bbox.height(); r++){
          // Compute the median for this pixel
          vals.clear();
          for (int i = 0; i < (int)tile_vec.size(); i++){
            ImageView<double> & tile_ref = tile_vec[i];
            double this_val = tile_ref(c, r);
            vals_all[i] = this_val; // Record the original order.
            if (this_val == m_opt.out_nodata_value)
              continue;
            vals.push_back(this_val);
          }
          if (vals.empty())
            continue;
          if (m_opt.median)
            tile(c, r) = math::destructive_median(vals);
          else
            tile(c, r) = math::destructive_nmad(vals);

          if (!m_opt.save_index_map)
            continue;
          // Record the index of the image that is closest to the
          // median value.  Note that the median can average two
          // values, so the median value may not equal exactly any of
          // the input values.
          double min_dist = std::numeric_limits<double>::max();
          for (size_t m = 0; m < vals_all.size(); m++) {
            double dist = fabs(vals_all[m] - tile(c, r));
            if (dist < min_dist) {
	      // Here we save the index not in the current array which
	      // is m, but in the full list of DEMs, some of which are
	      // likely skipped in this tile as they don't intersect
	      // it.
              index_map(c, r) = clip2dem_index[m];
              min_dist = dist;
            }
          }

        }// End row loop
      } // End col loop
    } // End median/nmad case

    // For max per block, find the sum of values in each DEM
    if (m_opt.block_max) {
      fill(tile, m_opt.out_nodata_value);
      int num_tiles = tile_vec.size();
      if (tile_vec.size() != dem_vec.size()) 
        vw_throw(ArgumentErr() << "Book-keeping error.\n");
      std::vector<double> tile_sum(num_tiles, 0);
      for (int i = 0; i < num_tiles; i++) {
        for (int c = 0; c < tile_vec[i].cols(); c++) {
          for (int r = 0; r < tile_vec[i].rows(); r++) {
            if (tile_vec[i](c, r) != m_opt.out_nodata_value) {
              tile_sum[i] += tile_vec[i](c, r);
            }
          }
        }
        // The whole purpose of --block-max is to print the sum of
        // pixels for each mapprojected image/DEM when doing SfS.
        // The documentation has a longer explanation.
        vw_out() << "\n" << bbox << " " << dem_vec[i]
                 << " pixel sum: " << tile_sum[i] << std::endl;
      }
      int max_index = std::distance(tile_sum.begin(),
                                    std::max_element(tile_sum.begin(), tile_sum.end()));
      if (max_index >= 0 && max_index < num_tiles) 
        tile = copy(tile_vec[max_index]);
    }

    // For priority blending length.
    if (use_priority_blend) {

      if (tile_vec.size() != weight_vec.size() || tile_vec.size() != clip2dem_index.size())
        vw_throw(ArgumentErr() << "There must be as many dem tiles as weight tiles.\n");

      // We will use the weights created so far only to burn holes in
      // the DEMs where we don't want blending. Then we will have to
      // recreate the weights. That because the current weights have
      // been interpolated from a different grid, and won't handle
      // erosion and bluring well.
      for (size_t clip_iter = 0; clip_iter < weight_vec.size(); clip_iter++) {
        for (int col = 0; col < weight_vec[clip_iter].cols(); col++){
          for (int row = 0; row < weight_vec[clip_iter].rows(); row++){
            if (weight_vec[clip_iter](col, row) <= 0)
              tile_vec[clip_iter](col, row) = m_opt.out_nodata_value;
          }
        }

        weight_vec[clip_iter] = grassfire(notnodata(tile_vec[clip_iter],
                                                    m_opt.out_nodata_value),
                                          m_opt.no_border_blend);
      }

      // Don't allow the weights to grow too fast, for uniqueness.
      for (size_t clip_iter = 0; clip_iter < weight_vec.size(); clip_iter++) {
        for (int col = 0; col < weight_vec[clip_iter].cols(); col++) {
          for (int row = 0; row < weight_vec[clip_iter].rows(); row++) {
            weight_vec[clip_iter](col, row)
              = std::min(weight_vec[clip_iter](col, row), double(m_bias));
          }
        }
      }

      // Blur the weights.
      for (size_t clip_iter = 0; clip_iter < weight_vec.size(); clip_iter++) {
        blur_weights(weight_vec[clip_iter], m_opt.weights_blur_sigma);
      }

      // Raise to power
      if (m_opt.weights_exp != 1) {
        for (size_t clip_iter = 0; clip_iter < weight_vec.size(); clip_iter++) {
          for (int col = 0; col < weight_vec[clip_iter].cols(); col++){
            for (int row = 0; row < weight_vec[clip_iter].rows(); row++){
              weight_vec[clip_iter](col, row)
                = pow(weight_vec[clip_iter](col, row), m_opt.weights_exp);
            }
          }
        }
      }

      // Now we are ready for blending
      fill(tile, m_opt.out_nodata_value);
      fill(weights, 0.0);

      if (m_opt.save_dem_weight >= 0)
        fill(saved_weight, 0.0);

      for (size_t clip_iter = 0; clip_iter < weight_vec.size(); clip_iter++) {
        for (int col = 0; col < weight_vec[clip_iter].cols(); col++){
          for (int row = 0; row < weight_vec[clip_iter].rows(); row++){

            double wt = weight_vec[clip_iter](col, row);
            if (wt <= 0)
              continue; // nothing to do

            // Initialize the tile
            if (tile(col, row) == m_opt.out_nodata_value)
              tile(col, row) = 0;

            tile(col, row)    += wt*tile_vec[clip_iter](col, row);
            weights(col, row) += wt;

            if (clip2dem_index[clip_iter] == m_opt.save_dem_weight)
              saved_weight(col, row) = wt;
          }
        }
      }

      // Compute the weighted average
      for (int col = 0; col < tile.cols(); col++){
        for (int row = 0; row < weights.rows(); row++){
          if (weights(col, row) > 0)
            tile(col, row) /= weights(col, row);

          if (m_opt.save_dem_weight >= 0 && weights(col, row) > 0)
            saved_weight(col, row) /= weights(col, row);

        }
      }

#if 0
  for (size_t clip_iter = 0; clip_iter < weight_vec.size(); clip_iter++) {
    // Dump the modifier weights
    GeoReference crop_georef = crop(m_out_georef, bbox);
    std::ostringstream os;
    os << "tile_weight_" << clip_iter << ".tif";
    vw_out() << "\nWriting: " << os.str() << std::endl;
    bool has_georef = true, has_nodata = true;
    block_write_gdal_image(os.str(), weight_vec[clip_iter],
                has_georef, crop_georef,
                has_nodata, -100,
                vw::GdalWriteOptions(),
                TerminalProgressCallback("asp", ""));
      }
#endif

    } // end considering the priority blending length

    // Save the weight instead
    if (m_opt.save_dem_weight >= 0)
      tile = saved_weight;

    // Save the index map instead
    if (m_opt.save_index_map)
      tile = index_map;

    // How many valid pixels are there in the tile
    long long int num_valid_in_tile = 0; // use int64 to not overlow for large images
    for (int col = 0; col < tile.cols(); col++) {
      for (int row = 0; row < tile.rows(); row++) {
        vw::Vector2 pix = vw::Vector2(col, row) + bbox.min();
        if (!orig_box.contains(pix))
          continue; // in case the box got expanded, ignore the padding
        if (tile(col, row) == m_opt.out_nodata_value)
          continue;
        num_valid_in_tile++;
      }
    }
    {
      // Lock and update the total number of valid pixels
      vw::Mutex::Lock lock(m_count_mutex);
      m_num_valid_pixels += num_valid_in_tile;
    }

    if (m_opt.first_dem_as_reference) {

      if (first_dem.cols() != tile.cols() || first_dem.rows() != tile.rows()) {
        vw_throw(ArgumentErr() << "Book-keeping error when blending into first DEM.\n");
      }

      // Wipe from the tile all values outside the perimeter of
      // first_dem. So we don't wipe values that happen to be
      // in the holes of first_dem.
      vw::ImageView<double> local_wts;
      bool fill_holes = true;
      centerline_weights(create_mask(first_dem, m_opt.out_nodata_value), local_wts,
                         BBox2(), fill_holes);
      for (int col = 0; col < tile.cols(); col++) {
        for (int row = 0; row < tile.rows(); row++) {
          if (local_wts(col, row) == 0)
            tile(col, row) = m_opt.out_nodata_value;
        }
      }
    }

    // Return the tile we created with fake borders to make it look
    // the size of the entire output image. So far we operated
    // on doubles, here we cast to RealT.
    return prerasterize_type(pixel_cast<RealT>(tile),
                             -bbox.min().x(), -bbox.min().y(),
                             cols(), rows());
  }

  template <class DestT>
  inline void rasterize(DestT const& dest, BBox2i bbox) const {
    vw::rasterize(prerasterize(bbox), dest, bbox);
  }
}; // End class DemMosaicView


/// Find the bounding box of all DEMs in the projected space.
/// - mosaic_bbox is the output bounding box in projected space
/// - dem_proj_bboxes and dem_pixel_bboxes are the locations of
///   each input DEM in the output DEM in projected and pixel coordinates.
void load_dem_bounding_boxes(Options       const& opt,
                             GeoReference  const& mosaic_georef,
                             BBox2              & mosaic_bbox, // Projected coordinates
                             std::vector<BBox2> & dem_proj_bboxes,
                             std::vector<BBox2i> & dem_pixel_bboxes) {

  vw_out() << "Determining the bounding boxes of the input DEMs.\n";

  // Initialize the outputs
  mosaic_bbox = BBox2();
  dem_proj_bboxes.clear();
  dem_pixel_bboxes.clear();
  
  TerminalProgressCallback tpc("", "\t--> ");
  tpc.report_progress(0);
  double inc_amount = 1.0 / double(opt.dem_files.size());

  BBox2 first_dem_proj_box;
  
  // Loop through all DEMs
  for (int dem_iter = 0; dem_iter < (int)opt.dem_files.size(); dem_iter++){ 

    // Open a handle to this DEM file
    DiskImageResourceGDAL in_rsrc(opt.dem_files[dem_iter]);
    DiskImageView<RealT>  img(opt.dem_files[dem_iter]);
    GeoReference          georef = read_georef(opt.dem_files[dem_iter]);
    BBox2i                pixel_box = bounding_box(img);

    dem_pixel_bboxes.push_back(pixel_box);

    if (dem_iter == 0) 
      first_dem_proj_box = georef.bounding_box(img);
    
    bool has_lonat = (georef.proj4_str().find("+proj=longlat") != std::string::npos ||
                      mosaic_georef.proj4_str().find("+proj=longlat") != std::string::npos);
    
    // Compute bounding box of this DEM. The simple case is when all DEMs have
    // the same projection, and it is not longlat, as then we need to worry about
    // a 360 degree shift.
    if ((!has_lonat) && mosaic_georef.overall_proj4_str() == georef.overall_proj4_str()){
      BBox2 proj_box = georef.bounding_box(img);
      mosaic_bbox.grow(proj_box);
      dem_proj_bboxes.push_back(proj_box);
    }else{
      // Compute the bounding box of the current image in projected
      // coordinates of the mosaic. There is always a worry that the
      // lonlat of the mosaic so far and of the current DEM will be
      // offset by 360 degrees. Try to deal with that.
      BBox2 proj_box;
      BBox2 imgbox = bounding_box(img);
      BBox2 mosaic_pixel_box;
      
      // Get the bbox of current mosaic in pixels.
      if (dem_iter == 0) {
        // TODO: Not robust. How to estimate the pixel extent of the
        // first DEM in the mosaic? Taking into account that
        // the first DEM lonlat box and the mosaic lonlat box
        // may be offset by 360 degrees?
        mosaic_pixel_box = imgbox;
      }else{
        mosaic_pixel_box = mosaic_georef.point_to_pixel_bbox(mosaic_bbox);
      }
      
      GeoTransform geotrans(georef, mosaic_georef, imgbox, mosaic_pixel_box);
      proj_box = geotrans.pixel_to_point_bbox(imgbox);

      // Notify the user if one of the input DEMS is going to wrap around the 
      // left and right sides of their output image.
      if ((dem_iter > 0) && geotrans.check_bbox_wraparound()) {
        vw_out() << "WARNING: Longitude wraparound detected from input DEM "
                 << opt.dem_files[dem_iter] << " to the output georeference. "
                 << "This can result in an output DEM *much* larger than expected. "
                 << "Consider changing your output georeference options or your inputs.\n";
      }

      mosaic_bbox.grow(proj_box);
      dem_proj_bboxes.push_back(proj_box);
    } // End second case

    tpc.report_incremental_progress(inc_amount);
  } // End loop through DEM files
  tpc.report_finished();

  // If the first dem is used as reference, no matter what use its own box
  if (opt.first_dem_as_reference) 
    mosaic_bbox = first_dem_proj_box;
  
} // End function load_dem_bounding_boxes


void handle_arguments(int argc, char *argv[], Options& opt) {

  po::options_description general_options("Options");
  general_options.add_options()
    ("dem-list-file,l", po::value<std::string>(&opt.dem_list_file),
	   "Text file listing the DEM files to mosaic, one per line.")
    ("output-prefix,o", po::value(&opt.out_prefix), "Specify the output prefix. One or more tiles will be written with this prefix. Alternatively, an exact output file can be specified, with a .tif extension.")
    ("tile-size",       po::value<int>(&opt.tile_size)->default_value(1000000),
	   "The maximum size of output DEM tile files to write, in pixels.")
    ("tile-index",      po::value<int>(&opt.tile_index),
     "The index of the tile to save (starting from zero). When this program is invoked, it will print out how many tiles are there. Default: save all tiles.")
    ("tile-list",      po::value(&opt.tile_list_str)->default_value(""),
     "List of tile indices (in quotes) to save. A tile index starts from 0.")
    ("priority-blending-length", po::value<int>(&opt.priority_blending_len)->default_value(0),
	   "If positive, keep unmodified values from the earliest available DEM except a band this wide measured in pixels inward of its boundary where blending with subsequent DEMs will happen.")
    ("no-border-blend", po::bool_switch(&opt.no_border_blend)->default_value(false),
	   "Only apply blending around holes, don't blend at image borders.  Not compatible with centerline weights.")
    ("tr",              po::value(&opt.tr),
	   "Output grid size, that is, the DEM resolution in target georeferenced units per pixel. Default: use the same resolution as the first DEM to be mosaicked.")
    ("t_srs",           po::value(&opt.target_srs_string)->default_value(""),
	   "Specify the output projection (PROJ.4 string). Default: use the one from the first DEM to be mosaicked.")
    ("t_projwin",       po::value(&opt.projwin),
     "Limit the mosaic to this region, with the corners given in georeferenced coordinates "
     "(xmin ymin xmax ymax). Max is exclusive. See the ``--tap`` option if desired to apply "
     "addition adjustments to this extent.")
    ("tap",  po::bool_switch(&opt.tap)->default_value(false),
     "Let the output grid be at integer multiples of the grid size (like "
     "the default behavior of point2dem and mapproject, and "
     "gdalwarp when invoked with -tap, though the latter does "
     "not have the half-a-pixel extra extent this tool has). If this "
     "option is not set, the input grids determine the output grid.")
    ("first",   po::bool_switch(&opt.first)->default_value(false),
	   "Keep the first encountered DEM value (in the input order).")
    ("last",    po::bool_switch(&opt.last)->default_value(false),
	   "Keep the last encountered DEM value (in the input order).")
    ("min",     po::bool_switch(&opt.min)->default_value(false),
	   "Keep the smallest encountered DEM value.")
    ("max",     po::bool_switch(&opt.max)->default_value(false),
	   "Keep the largest encountered DEM value.")
    ("mean",    po::bool_switch(&opt.mean)->default_value(false),
	   "Find the mean DEM value.")
    ("stddev",    po::bool_switch(&opt.stddev)->default_value(false),
	   "Find the standard deviation of the DEM values.")
    ("median",  po::bool_switch(&opt.median)->default_value(false),
	   "Find the median DEM value (this can be memory-intensive, fewer threads are suggested).")
    ("nmad",  po::bool_switch(&opt.nmad)->default_value(false),
	   "Find the normalized median absolute deviation DEM value (this can be memory-intensive, fewer threads are suggested).")
    ("count",   po::bool_switch(&opt.count)->default_value(false),
     "Each pixel is set to the number of valid DEM heights at that pixel.")
    ("hole-fill-length",   po::value(&opt.hole_fill_len)->default_value(0),
	   "Maximum dimensions of a hole in the DEM to fill, in pixels. See also --fill-search-radius.")
     ("fill-search-radius",   po::value(&opt.fill_search_radius)->default_value(0.0),
      "Fill an invalid pixel with a weighted average of pixel values within this radius in pixels. The weight is 1/(factor * dist^power + 1), where the distance is measured in pixels. See an example in the doc. See also --fill-power, --fill-percent and --fill-num-passes.")
      ("fill-power", po::value(&opt.fill_power)->default_value(8.0),
      "Power exponent to use when filling nodata values with --fill-search-radius.")
      ("fill-percent", po::value(&opt.fill_percent)->default_value(10.0),
      "Fill an invalid pixel using weighted values of neighbors only if the percentage of valid pixels within the radius given by --fill-search-radius is at least this")
      ("fill-num-passes", po::value(&opt.fill_num_passes)->default_value(0),
      "Fill invalid values using --fill-search-radius this many times.")
    ("erode-length",    po::value<int>(&opt.erode_len)->default_value(0),
	   "Erode the DEM by this many pixels at boundary.")
    ("block-max", po::bool_switch(&opt.block_max)->default_value(false),
     "For each block of size --block-size, keep the DEM with the largest sum of values in the block.")
    ("georef-tile-size",    po::value<double>(&opt.geo_tile_size),
     "Set the tile size in georeferenced (projected) units (e.g., degrees or meters).")
    ("output-nodata-value", po::value<double>(&opt.out_nodata_value),
     "No-data value to use on output. Default: use the one from the first DEM to be mosaicked.")
    ("ot",  po::value(&opt.output_type)->default_value("Float32"), "Output data type. Supported types: Byte, UInt16, Int16, UInt32, Int32, Float32. If the output type is a kind of integer, values are rounded and then clamped to the limits of that type.")
    ("weights-blur-sigma", po::value<double>(&opt.weights_blur_sigma)->default_value(5.0),
     "The standard deviation of the Gaussian used to blur the weights. Higher value results in smoother weights and blending. Set to 0 to not use blurring.")
    ("weights-exponent",   po::value<double>(&opt.weights_exp)->default_value(2.0),
     "The weights used to blend the DEMs should increase away from the boundary as a power with this exponent. Higher values will result in smoother but faster-growing weights.")
    ("use-centerline-weights",   po::bool_switch(&opt.use_centerline_weights)->default_value(false),
     "Compute weights based on a DEM centerline algorithm. Produces smoother weights if the input DEMs don't have holes or complicated boundary.")
    ("dem-blur-sigma", po::value<double>(&opt.dem_blur_sigma)->default_value(0.0),
     "Blur the DEM using a Gaussian with this value of sigma. A larger value will blur more. Default: No blur.")
    ("nodata-threshold", po::value(&opt.nodata_threshold)->default_value(std::numeric_limits<double>::quiet_NaN()),
     "Values no larger than this number will be interpreted as no-data.")
    ("propagate-nodata", po::bool_switch(&opt.propagate_nodata)->default_value(false),
     "Set a pixel to nodata if any input DEM is also nodata at that location.")
    ("extra-crop-length", po::value<int>(&opt.extra_crop_len)->default_value(200),
     "Crop the DEMs this far from the current tile (measured in pixels) before blending them (a small value may result in artifacts).")
    ("block-size",      po::value<int>(&opt.block_size)->default_value(0), "A large value can result in increased memory usage.")
    ("save-dem-weight",      po::value<int>(&opt.save_dem_weight),
     "Save the weight image that tracks how much the input DEM with given index contributed to the output mosaic at each pixel (smallest index is 0).")
    ("first-dem-as-reference", po::bool_switch(&opt.first_dem_as_reference)->default_value(false),
     "The output DEM will have the same size, grid, and georeference as the first one, with the other DEMs blended within its perimeter.")
    ("this-dem-as-reference", po::value(&opt.this_dem_as_reference)->default_value(""),
     "The output DEM will have the same size, grid, and georeference as this one, but it will not be used in the mosaic.")
    ("force-projwin", po::bool_switch(&opt.force_projwin)->default_value(false),
     "Make the output mosaic fill precisely the specified projwin, by padding it if necessary and aligning the output grid to the region.")
    ("save-index-map",   po::bool_switch(&opt.save_index_map)->default_value(false),
     "For each output pixel, save the index of the input DEM it came from (applicable only for --first, --last, --min, --max, --median, and --nmad). A text file with the index assigned to each input DEM is saved as well.");

  // Use in GdalWriteOptions '--tif-tile-size' rather than '--tile-size', to not conflict
  // with the '--tile-size' definition used by this tool.
  bool adjust_tile_size_opt = true; 
  general_options.add(vw::GdalWriteOptionsDescription(opt, adjust_tile_size_opt));
  
  po::options_description positional("");
  po::positional_options_description positional_desc;

  std::string usage("[options] <dem files or -l dem_files_list.txt> -o output_file_prefix");
  bool allow_unregistered = true;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                             positional, positional_desc, usage,
                             allow_unregistered, unregistered);

  // Error checking
  if (opt.out_prefix == "")
    vw_throw(ArgumentErr() << "No output prefix was specified.\n"
                           << usage << general_options);
  if (opt.num_threads == 0)
    vw_throw(ArgumentErr() << "The number of threads must be set and positive.\n"
                           << usage << general_options);
  if (opt.erode_len < 0)
    vw_throw(ArgumentErr() << "The erode length must not be negative.\n"
                           << usage << general_options);
  if (opt.extra_crop_len < 0)
    vw_throw(ArgumentErr() << "The blending length must not be negative.\n"
                           << usage << general_options);
  if (opt.hole_fill_len < 0)
    vw_throw(ArgumentErr() << "The hole fill length must not be negative.\n"
                           << usage << general_options);
  if (opt.fill_search_radius < 0.0)
    vw_throw(ArgumentErr() << "The fill search radius must be non-negative.\n"
                           << usage << general_options);
  if (opt.fill_power < 0.0)
    vw_throw(ArgumentErr() << "The fill factor must be non-negative.\n"
                           << usage << general_options);
    if (opt.fill_percent <= 0.0 || opt.fill_percent > 100.0)
    vw_throw(ArgumentErr() << "The fill percent must be in the range (0, 100].\n"
                           << usage << general_options);
  if (opt.fill_num_passes < 0)
    vw_throw(ArgumentErr() << "The number of fill passes must not be negative.\n"
                           << usage << general_options);
  if (opt.tile_size <= 0)
    vw_throw(ArgumentErr() << "The size of a tile in pixels must be positive.\n"
                           << usage << general_options);
  if (opt.priority_blending_len < 0)
    vw_throw(ArgumentErr() << "The priority blending length must not be negative.\n"
                           << usage << general_options);

  // If priority blending is used, need to adjust extra_crop_len accordingly
  opt.extra_crop_len = std::max(opt.extra_crop_len, 3*opt.priority_blending_len);

  // Make sure no more than one of these options is enabled.
  int noblend = no_blend(opt);
  if (noblend > 1)
    vw_throw(ArgumentErr() << "At most one of the options --first, --last, "
         << "--min, --max, -mean, --stddev, --median, --nmad, --count can be specified.\n"
         << usage << general_options);

  if (opt.geo_tile_size < 0)
    vw_throw(ArgumentErr() << "The size of a tile in georeferenced units must not be negative.\n"
                           << usage << general_options);

  if (noblend && opt.priority_blending_len > 0) {
    vw_throw(ArgumentErr()
       << "Priority blending cannot happen if any of the statistics DEMs are computed.\n"
       << usage << general_options);
  }

  if (opt.priority_blending_len > 0 && opt.weights_exp == 2) {
    vw_out() << "Increasing --weights-exponent to 3 for smoother blending.\n";
    opt.weights_exp = 3;
  }
  
  if (noblend && !opt.first && !opt.last && !opt.min && !opt.max && !opt.mean
      && opt.save_dem_weight >= 0) {
    vw_throw(ArgumentErr() << "Cannot save the weights unless blending is on or one of "
                           << "--first, --last, --min, --max, --mean is invoked.\n"
                           << usage << general_options);
  }

  if (opt.save_index_map && !opt.first && !opt.last &&
                            !opt.min && !opt.max && !opt.median && !opt.nmad)
    vw_throw(ArgumentErr() << "Cannot save an index map unless one of "
                           << "--first, --last, --min, --max, --median, --nmad is invoked.\n"
                           << usage << general_options);

  if (opt.save_dem_weight >= 0 && opt.save_index_map)
    vw_throw(ArgumentErr()
       << "Cannot save both the index map and the DEM weights at the same time.\n"
       << usage << general_options);

  // For compatibility with the GDAL tools, allow the min and max to be reversed.
  if (opt.projwin != BBox2()) {
    if (opt.projwin.min().x() > opt.projwin.max().x())
      std::swap(opt.projwin.min().x(), opt.projwin.max().x());
    if (opt.projwin.min().y() > opt.projwin.max().y())
      std::swap(opt.projwin.min().y(), opt.projwin.max().y());
  }
  
  if (opt.weights_blur_sigma < 0.0)
    vw_throw(ArgumentErr() << "The value --weights-blur-sigma must be non-negative.\n"
             << usage << general_options);
  
  if (opt.dem_blur_sigma < 0.0)
    vw_throw(ArgumentErr() << "The value --dem-blur-sigma must be non-negative.\n"
             << usage << general_options);

  if (opt.weights_exp <= 0)
    vw_throw(ArgumentErr() << "The weights exponent must be positive.\n"
             << usage << general_options);

  if (opt.priority_blending_len > 0 && opt.use_centerline_weights)
    vw_throw(ArgumentErr() << "The --priority-blending-length and --use-centerline-weights options cannot be used together, as the latter expects no holes in the DEM, but the priority blending length works by internally hollowing out the non-priority DEMs before blending.\n"
             << usage << general_options);

  // Read the DEMs
  if (opt.dem_list_file != ""){ // Get them from a list

    if (!unregistered.empty())
      vw_throw(ArgumentErr() << "The DEMs were specified via a list. "
			     << "There were however extraneous files or options passed in.\n"
			     << usage << general_options);

    std::ifstream is(opt.dem_list_file.c_str());
    std::string file;
    while (is >> file)
      opt.dem_files.push_back(file);
    if (opt.dem_files.empty())
      vw_throw(ArgumentErr() << "No DEM files to mosaic.\n");
    is.close();

  }else{  // Get them from the command line

    if (unregistered.empty())
      vw_throw(ArgumentErr() << "No input DEMs were specified.\n"
			     << usage << general_options);
    opt.dem_files = unregistered;
  }

  if (opt.this_dem_as_reference != "" && opt.first_dem_as_reference) {
    vw_throw(ArgumentErr() << "Cannot have both options --first-dem-as-reference "
             << "and --this-dem-as-reference.\n"
	     << usage << general_options);
  }

  // We will co-opt the logic of first_dem_as_reference but won't blend the reference DEM
  if (opt.this_dem_as_reference != "") {
    opt.first_dem_as_reference = true;
    opt.dem_files.insert(opt.dem_files.begin(), opt.this_dem_as_reference);
  }
  
  if (int(opt.dem_files.size()) <= opt.save_dem_weight) {
    vw_throw(ArgumentErr() << "Cannot save weights for given index as it is out of bounds.\n"
	     << usage << general_options);
  }

  // When too many things should be done at the same time it is tricky
  // to have them work correctly. So prohibit that. Let the user run
  // one operation at a time.
  int num_ops = (opt.dem_blur_sigma > 0) + (opt.hole_fill_len > 0) +
                (opt.fill_search_radius > 0) + (opt.erode_len > 0);
  if (num_ops > 1) 
    vw_throw(ArgumentErr() << "Cannot fill holes (based on size or search radius), blur, "
                           << "and erode the input DEM at the same time.\n" 
                           << usage << general_options);
  
  if (num_ops > 0 && 
      (opt.target_srs_string != "" || opt.tr > 0 || opt.dem_files.size() > 1 ||
       opt.priority_blending_len > 0))
    vw::vw_throw(vw::ArgumentErr() << "Cannot fill holes (based on size or search radius), "
       << "blur, or erode, if there is more than one input DEM, or reprojection, "
       << "or priority blending is desired. These operations should be done "
       << "one at a time as there may be issues due to the fact each input DEM has "
       << "its own grid size and also the order of operations.\n"
       << usage << general_options);

  // Create the output directory
  vw::create_out_dir(opt.out_prefix);

  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);

  if (!vm.count("output-nodata-value")) {
    // Set a default out_nodata_value, but remember that this is
    // set internally, not by the user.
    opt.has_out_nodata = false;
    opt.out_nodata_value = -std::numeric_limits<RealT>::max();
  }else
    opt.has_out_nodata = true;

  // Cast this to float. All our nodata are float.
  opt.nodata_threshold = RealT(opt.nodata_threshold);

  // Parse the list of tiles to save. First replace commas and semicolons by a space.
  std::replace(opt.tile_list_str.begin(), opt.tile_list_str.end(), ',', ' ');
  std::replace(opt.tile_list_str.begin(), opt.tile_list_str.end(), ';', ' ');
  opt.tile_list.clear();
  std::istringstream os(opt.tile_list_str);
  int val;
  while (os >> val){
    opt.tile_list.insert(val);
  }
  
} // End function handle_arguments

int main(int argc, char *argv[]) {

  Options opt;
  
  try {

    handle_arguments(argc, argv, opt);

    // TODO: Fix here. If the DEM is double, read the nodata as double,
    // without casting to float. If it is float, cast to float.
    
    // Read nodata from first DEM, unless the user chooses to specify it.
    if (!opt.has_out_nodata){
      DiskImageResourceGDAL in_rsrc(opt.dem_files[0]);
      // Since the DEMs have float pixels, we must read the no-data as
      // float as well. (this is a bug fix). Yet we store it in a
      // double, as we will cast the DEM pixels to double as well.
      if (in_rsrc.has_nodata_read()) opt.out_nodata_value = RealT(in_rsrc.nodata_read());
    }

    // Watch for underflow, if mixing doubles and float. Particularly problematic
    // is when the nodata_value cannot be represented exactly as a float.
    if (opt.out_nodata_value < static_cast<double>(-std::numeric_limits<RealT>::max()) ||
        RealT(opt.out_nodata_value)  != double(opt.out_nodata_value)) {
      vw_out() << "The no-data value cannot be represented exactly as a float. "
	       << "Changing it to the smallest float.\n";
      opt.out_nodata_value = static_cast<double>(-std::numeric_limits<RealT>::max());
    }

    vw_out() << "Using output no-data value: " << opt.out_nodata_value << std::endl;

    // Form the mosaic georef. The georef of the first DEM is used as
    // initial guess unless user wants to change the resolution and projection.
    if (opt.target_srs_string != "")
      opt.target_srs_string = processed_proj4(opt.target_srs_string);

    // By default the output georef is equal to the first input georef
    GeoReference mosaic_georef = read_georef(opt.dem_files[0]);

    if (opt.first_dem_as_reference) {
      if (opt.target_srs_string != "" || opt.tr > 0 || opt.projwin != BBox2()) 
        vw_throw(ArgumentErr()
                 << "Cannot change the projection, spacing, or output box, if the first DEM "
                 << "is to be used as reference.\n");
      if (opt.first  || opt.last || opt.min    || opt.max || opt.mean || 
          opt.median || opt.nmad || opt.stddev ||
          opt.priority_blending_len > 0 || //opt.save_dem_weight >= 0 ||
          !boost::math::isnan(opt.nodata_threshold)) {
        vw_throw(ArgumentErr()
                 << "Cannot do anything except regular blending if the first DEM "
                 << "is to be used as reference.\n");
      }
    }
  
    double spacing = opt.tr;
    if (opt.target_srs_string != ""                                              &&
      opt.target_srs_string != processed_proj4(mosaic_georef.overall_proj4_str()) &&
      spacing <= 0){
        vw_throw(ArgumentErr()
           << "Changing the projection was requested. The output DEM "
           << "resolution must be specified via the --tr option.\n");
    }

    if (opt.target_srs_string != ""){
      // Set the srs string into georef.
      bool have_user_datum = false, have_input_georef = false;
      Datum user_datum;
      asp::set_srs_string(opt.target_srs_string,
			  have_user_datum, user_datum,
                          have_input_georef, mosaic_georef);
    }

    // Steal the datum and its name from the input, if the output
    // datum name is unknown.
    if (mosaic_georef.datum().name() == "unknown"){
      GeoReference georef = read_georef(opt.dem_files[0]);
      if (mosaic_georef.datum().semi_major_axis() == georef.datum().semi_major_axis() &&
    	  mosaic_georef.datum().semi_minor_axis() == georef.datum().semi_minor_axis()){
          vw_out() << "Using the datum: " << georef.datum() << std::endl;
          mosaic_georef.set_datum(georef.datum());
      }
    }

    // Use desired spacing if user-specified
    if (spacing > 0.0){
      // Get lonlat bounding box of the first DEM.
      DiskImageView<RealT> dem0(opt.dem_files[0]);
      BBox2 llbox0 = mosaic_georef.pixel_to_lonlat_bbox(bounding_box(dem0));
      
      // Reset transform with user provided spacing.
      Matrix<double,3,3> transform = mosaic_georef.transform();
      transform.set_identity();
      transform(0, 0) =  spacing;
      transform(1, 1) = -spacing;
      mosaic_georef.set_transform(transform);

      // Set the translation part of the transform so that the origin
      // maps to the lonlat box corner. This is still not fully
      // reliable, but better than nothing. We will adjust
      // mosaic_georef later on.
      vw::Vector2 ul = mosaic_georef.lonlat_to_pixel(Vector2(llbox0.min().x(), llbox0.max().y()));
      mosaic_georef = crop(mosaic_georef, ul.x(), ul.y());

    }else {
      // Update spacing variable from the current transform
      spacing = mosaic_georef.transform()(0, 0);
    }
    
    // if the user specified the tile size in georeferenced units.
    if (opt.geo_tile_size > 0){
      opt.tile_size = (int)round(opt.geo_tile_size/spacing);
      vw_out() << "Tile size in pixels: " << opt.tile_size << "\n";
    }
    opt.tile_size = std::max(opt.tile_size, 1);

    // Load the bounding boxes from all of the DEMs
    vw::BBox2 mosaic_bbox;
    std::vector<BBox2> dem_proj_bboxes;
    std::vector<BBox2i> dem_pixel_bboxes, loaded_dem_pixel_bboxes;
    load_dem_bounding_boxes(opt, mosaic_georef, mosaic_bbox,
                            dem_proj_bboxes, dem_pixel_bboxes);


    if (opt.tap) {
      // Ensure that the grid is at integer multiples of grid size
      mosaic_bbox.min() = spacing * floor(mosaic_bbox.min() / spacing);
      mosaic_bbox.max() = spacing * ceil(mosaic_bbox.max()  / spacing);
      if (opt.projwin != BBox2()) {
        opt.projwin.min() = spacing * floor(opt.projwin.min() / spacing);
        opt.projwin.max() = spacing * ceil(opt.projwin.max()  / spacing);
      }
    }
    
    if (opt.projwin != BBox2()) {
      // If to create the mosaic only in a given region
      mosaic_bbox.crop(opt.projwin);
      // Crop the proj boxes as well
      for (int dem_iter = 0; dem_iter < (int)opt.dem_files.size(); dem_iter++)
        dem_proj_bboxes[dem_iter].crop(opt.projwin);
      if (opt.force_projwin) 
        mosaic_bbox = opt.projwin;
    }
    
    // First-guess pixel box
    BBox2 pixel_box = custom_point_to_pixel_bbox(mosaic_georef, mosaic_bbox);

    // Take care of numerical artifacts
    vw::Vector2 beg_pix = pixel_box.min();
    if (norm_2(beg_pix - round(beg_pix)) < g_tol)
      beg_pix = round(beg_pix);
    mosaic_georef = crop(mosaic_georef, beg_pix[0], beg_pix[1]);

    // Image size
    pixel_box = custom_point_to_pixel_bbox(mosaic_georef, mosaic_bbox);
    vw::Vector2 end_pix = pixel_box.max();
    int cols = (int)round(end_pix[0]); // end_pix is the last pix in the image
    int rows = (int)round(end_pix[1]);

    // Form the mosaic and write it to disk
    vw_out()<< "The size of the mosaic is " << cols << " x " << rows << " pixels.\n";
    vw_out()<< "The output georeference is\n" << mosaic_georef << std::endl;

    // This bias is very important. This is how much we should read from
    // the images beyond the current boundary to avoid tiling artifacts.
    // The +1 is to ensure extra pixels beyond the hole fill length.
    int bias = opt.erode_len + opt.extra_crop_len + opt.hole_fill_len
      + opt.fill_search_radius
      + 2*std::max(vw::compute_kernel_size(opt.weights_blur_sigma),
                   vw::compute_kernel_size(opt.dem_blur_sigma))
                   + 1;

    // The next power of 2 >= 4*bias. We want to make the blocks big,
    // to reduce overhead from this bias, but not so big that it may
    // not fit in memory.
    int block_size = nextpow2(4.0*bias);
    block_size = std::max(block_size, 256); // don't make them too small though
    if (opt.block_size > 0)
      block_size = opt.block_size;

    // See if to lump all mosaic in just a given file, rather than creating tiles.
    bool write_to_precise_file = (opt.out_prefix.size() >= 4 &&
				   opt.out_prefix.substr(opt.out_prefix.size()-4, 4) == ".tif");
      
    int num_tiles_x = (int)ceil((double)cols/double(opt.tile_size));
    int num_tiles_y = (int)ceil((double)rows/double(opt.tile_size));
    if (num_tiles_x <= 0) num_tiles_x = 1;
    if (num_tiles_y <= 0) num_tiles_y = 1;
    int num_tiles = num_tiles_x*num_tiles_y;
    vw_out() << "Number of tiles: " << num_tiles_x << " x "
             << num_tiles_y << " = " << num_tiles << ".\n";
    
    if (opt.tile_index >= num_tiles){
      vw_out() << "Tile with index: " << opt.tile_index
	             << " is out of bounds." << std::endl;
      return 0;
    }

    if (num_tiles > 1 && write_to_precise_file) 
      vw_throw(ArgumentErr() << "Cannot fit all mosaic in the given output file name. "
	       << "Hence specify an output prefix instead, and then multiple "
	       << "tiles will be created.\n");
    
    // If to use a range
    if (!opt.tile_list.empty() && opt.tile_index >= 0) 
      vw_throw(ArgumentErr() << "Cannot specify both tile index and tile range.\n");

    // See if to save all tiles, or an individual tile.
    int start_tile = opt.tile_index, end_tile = opt.tile_index + 1;
    if (opt.tile_index < 0){
      start_tile = 0;
      end_tile = num_tiles;
    }

    // Compute the bounding box of each output tile
    std::vector<BBox2i> tile_pixel_bboxes;
    for (int tile_id = start_tile; tile_id < end_tile; tile_id++){

      int tile_index_y = tile_id / num_tiles_x;
      int tile_index_x = tile_id - tile_index_y*num_tiles_x;
      BBox2i tile_box(tile_index_x*opt.tile_size,
		      tile_index_y*opt.tile_size,
		      opt.tile_size, opt.tile_size);

      // Bounding box of this tile in pixels in the output image
      tile_box.crop(BBox2i(0, 0, cols, rows));

      tile_pixel_bboxes.push_back(tile_box);
    }

    // Store the no-data values, pointers to images, and georeferences (for speed).
    vw_out() << "Reading the input DEMs.\n";
    std::vector<double>       nodata_values;
    std::vector<GeoReference> georefs;
    std::vector<std::string>  loaded_dems;
    DiskImageManager<RealT>   imgMgr;

    BBox2i output_dem_box = BBox2i(0, 0, cols, rows); // output DEM box
    
    // Loop through all DEMs
    for (int dem_iter = 0; dem_iter < (int)opt.dem_files.size(); dem_iter++){

      // Get the DEM bounding box that we previously computed (output projected coords)
      BBox2 dem_bbox = dem_proj_bboxes[dem_iter];

      // Go through each of the tile bounding boxes and see they intersect this DEM
      bool use_this_dem = false;
      for (int tile_id = start_tile; tile_id < end_tile; tile_id++){

        if (!opt.tile_list.empty() && opt.tile_list.find(tile_id) == opt.tile_list.end()) 
          continue;
        
        // Get tile bbox in pixels, then convert it to projected coords.
        BBox2i tile_pixel_box = tile_pixel_bboxes[tile_id - start_tile];
        BBox2  tile_proj_box  = mosaic_georef.pixel_to_point_bbox(tile_pixel_box);

        if (tile_proj_box.intersects(dem_bbox)) {
          use_this_dem = true;
          break;
        }
      }
      if (use_this_dem == false)
        continue; // Skip to the next DEM if we don't need this one.

      // The GeoTransform will hide the messy details of conversions
      // from pixels to points and lon-lat.
      GeoReference georef  = read_georef(opt.dem_files[dem_iter]);
      BBox2i dem_pixel_box = dem_pixel_bboxes[dem_iter];
      GeoTransform geotrans(georef, mosaic_georef, dem_pixel_box, output_dem_box);

      // Get the current DEM bounding box in pixel units of the output mosaicked DEM
      BBox2 curr_box = geotrans.forward_bbox(dem_pixel_box);
      curr_box.crop(output_dem_box);

      // This is a fix for GDAL crashing when there are too many open
      // file handles. In such situation, just selectively close the
      // handles furthest from the current location.
      imgMgr.add_file_handle_not_thread_safe(opt.dem_files[dem_iter], curr_box);
      
      double curr_nodata_value = opt.out_nodata_value;
      try {
        // Get the nodata-value. Need a try block, in case we can't
        // open more handles.
        DiskImageResourceGDAL in_rsrc(opt.dem_files[dem_iter]);
        if (in_rsrc.has_nodata_read())
          curr_nodata_value = RealT(in_rsrc.nodata_read());
      }catch(std::exception const& e){
        // Try again
        imgMgr.freeup_handles_not_thread_safe();
        DiskImageResourceGDAL in_rsrc(opt.dem_files[dem_iter]);
        if (in_rsrc.has_nodata_read())
          curr_nodata_value = RealT(in_rsrc.nodata_read());
      }
      
      loaded_dems.push_back(opt.dem_files[dem_iter]);

      if (!boost::math::isnan(opt.nodata_threshold)) 
        curr_nodata_value = opt.nodata_threshold;
      
      // Add the info for this DEM to the appropriate vectors
      nodata_values.push_back(curr_nodata_value);
      georefs.push_back(georef);
      loaded_dem_pixel_bboxes.push_back(dem_pixel_box);
    } // End loop through DEM files

    // If there are 17 tiles, let them be tile-00, ..., tile-16.
    int num_digits = 1;
    int tens = 10;
    while (num_tiles - 1 >= tens){
      num_digits++;
      tens *= 10;
    }
    
    // Time to generate each of the output tiles
    for (int tile_id = start_tile; tile_id < end_tile; tile_id++){

      if (!opt.tile_list.empty() && opt.tile_list.find(tile_id) == opt.tile_list.end()) 
        continue;
      
      // Get the bounding box we previously computed
      vw::BBox2i tile_box = tile_pixel_bboxes[tile_id - start_tile];

      std::string dem_tile;
      if (!write_to_precise_file) {
        std::ostringstream os;
        os << opt.out_prefix << "-tile-"
            << std::setfill('0') << std::setw(num_digits) << tile_id
            << tile_suffix(opt) << ".tif";
        dem_tile = os.str();
      } else {
        dem_tile = opt.out_prefix; // the file name was set by user
      }
      
      // Set up tile image and metadata
      long long int num_valid_pixels; // Will be populated when saving to disk
      vw::Mutex count_mutex; // to lock when updating num_valid_pixels

      ImageViewRef<RealT> out_dem
        = crop(DemMosaicView(cols, rows, bias, opt,
                             imgMgr, georefs,
                             mosaic_georef, nodata_values,
                             loaded_dem_pixel_bboxes,
                             num_valid_pixels, count_mutex),
               tile_box);
      GeoReference crop_georef = crop(mosaic_georef, tile_box.min().x(),
				      tile_box.min().y());

      // Raster the tile to disk. Optionally cast to int (may be
      // useful for mosaicking ortho images).
      vw_out() << "Writing: " << dem_tile << std::endl;
      bool has_georef = true, has_nodata = true;
      TerminalProgressCallback tpc("asp", "\t--> ");
      if (opt.output_type == "Float32") 
        asp::save_with_temp_big_blocks(block_size, dem_tile, out_dem,
                                       has_georef, crop_georef,
                                       has_nodata, opt.out_nodata_value, opt, tpc);
      else if (opt.output_type == "Byte") 
        asp::save_with_temp_big_blocks(block_size, dem_tile,
				       per_pixel_filter(out_dem, RoundAndClamp<uint8, RealT>()),
                                       has_georef, crop_georef,
                                       has_nodata, vw::round_and_clamp<uint8>(opt.out_nodata_value),
                                       opt, tpc);
      else if (opt.output_type == "UInt16") 
        asp::save_with_temp_big_blocks(block_size, dem_tile,
				       per_pixel_filter(out_dem, RoundAndClamp<uint16, RealT>()),
                                       has_georef, crop_georef,
                                       has_nodata, vw::round_and_clamp<uint16>(opt.out_nodata_value),
                                       opt, tpc);
      else if (opt.output_type == "Int16") 
        asp::save_with_temp_big_blocks(block_size, dem_tile,
				       per_pixel_filter(out_dem, RoundAndClamp<int16, RealT>()),
                                       has_georef, crop_georef,
                                       has_nodata, vw::round_and_clamp<int16>(opt.out_nodata_value),
                                       opt, tpc);
      else if (opt.output_type == "UInt32") 
        asp::save_with_temp_big_blocks(block_size, dem_tile,
				       per_pixel_filter(out_dem, RoundAndClamp<uint32, RealT>()),
                                       has_georef, crop_georef,
                                       has_nodata, vw::round_and_clamp<uint32>(opt.out_nodata_value),
                                       opt, tpc);
      else if (opt.output_type == "Int32") 
        asp::save_with_temp_big_blocks(block_size, dem_tile,
				       per_pixel_filter(out_dem, RoundAndClamp<int32, RealT>()),
                                       has_georef, crop_georef,
                                       has_nodata, vw::round_and_clamp<int32>(opt.out_nodata_value),
                                       opt, tpc);
      else
        vw_throw(NoImplErr() << "Unsupported output type: " << opt.output_type << ".\n");

      vw_out() << "Number of valid (not no-data) pixels written: " << num_valid_pixels
               << "."<< std::endl;
      if (num_valid_pixels == 0) {
        vw_out() << "Removing tile with no valid pixels: " << dem_tile << std::endl;
        boost::filesystem::remove(dem_tile);
      }
      
    } // End loop through tiles

    // Write the name of each DEM file that was used together with its index
    if (opt.save_index_map) {
      std::string index_map = opt.out_prefix + "-index-map.txt";
      vw_out() << "Writing: " << index_map << std::endl;
      std::ofstream ih(index_map.c_str());
      for (int dem_iter = 0; dem_iter < (int)loaded_dems.size(); dem_iter++){
        ih << opt.dem_files[dem_iter] << ' ' << dem_iter << std::endl;
      }
    }

  } ASP_STANDARD_CATCHES;

  return 0;
}
