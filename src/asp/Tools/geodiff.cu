#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <asp/Core/PointUtils.h>
#include <vw/FileIO/DiskImageView.h>
#include <vw/Cartography/GeoTransform.h>
#include <vw/Cartography/PointImageManipulation.h>


using std::endl;
using std::string;

using namespace vw;
using namespace vw::cartography;

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
namespace po = boost::program_options;
namespace fs = boost::filesystem;

// Function to convert a masked geodetic vector to a masked altitude vector
class MGeodeticToMAltitude : public ReturnFixedType<PixelMask<double>> {
public:
  PixelMask<double> operator()(PixelMask<Vector3> const& v) const {
    if (!is_valid(v)) {
      return PixelMask<double>();
    }
    return PixelMask<double>(v.child()[2]);
  }
};

// Function to mask NaN pixels 
class MaskNaN : public ReturnFixedType<PixelMask<double>> {
public:
  PixelMask<double> operator()(PixelMask<double> const& v) const {
    if (!is_valid(v) || std::isnan(v.child())) {
      return PixelMask<double>();
    }
    return v;
  }
};

struct Options : vw::GdalWriteOptions {
  string dem1_file, dem2_file, output_prefix, csv_format_str, csv_proj4_str;
  double nodata_value;

  bool use_float, use_absolute;
};

void handle_arguments(int argc, char *argv[], Options& opt) {
  po::options_description general_options("");
  general_options.add_options()
    ("nodata-value",    po::value(&opt.nodata_value)->default_value(-32768),      
                        "The no-data value to use, unless present in the DEM geoheaders.")
    ("output-prefix,o", po::value(&opt.output_prefix),                            
                        "Specify the output prefix.")
    ("float",           po::bool_switch(&opt.use_float)->default_value(false),    
                        "Output using float (32 bit) instead of using doubles (64 bit).")
    ("absolute",        po::bool_switch(&opt.use_absolute)->default_value(false), 
     "Output the absolute difference as opposed to just the difference.")
    ("csv-format",     po::value(&opt.csv_format_str)->default_value(""),
     asp::csv_opt_caption().c_str())
    ("csv-proj4",      po::value(&opt.csv_proj4_str)->default_value(""), "The PROJ.4 string to use to interpret the entries in input CSV file. If not specified, it will be borrowed from the DEM.");
  general_options.add(vw::GdalWriteOptionsDescription(opt));

  po::options_description positional("");
  positional.add_options()
    ("dem1", po::value(&opt.dem1_file), "Explicitly specify the first dem")
    ("dem2", po::value(&opt.dem2_file), "Explicitly specify the second dem");

  po::positional_options_description positional_desc;
  positional_desc.add("dem1", 1);
  positional_desc.add("dem2", 1);

  std::string usage("[options] <dem1> <dem2>");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                             positional, positional_desc, usage,
                             allow_unregistered, unregistered);

  if (opt.dem1_file.empty() || opt.dem2_file.empty())
    vw_throw(ArgumentErr() << "Requires <dem1> and <dem2> in order to proceed.\n\n"
             << usage << general_options);

  if (opt.output_prefix.empty()) {
    opt.output_prefix = fs::basename(opt.dem1_file) + "__" + fs::basename(opt.dem2_file);
  }

  vw::create_out_dir(opt.output_prefix);
}

void georef_sanity_checks(GeoReference const& georef1, GeoReference const& georef2){

  // We don't support datum changes!
  if (std::abs(georef1.datum().semi_major_axis()
               - georef2.datum().semi_major_axis()) > 0.1 ||
      std::abs(georef1.datum().semi_minor_axis()
               - georef2.datum().semi_minor_axis()) > 0.1 ||
      georef1.datum().meridian_offset() != georef2.datum().meridian_offset()) {
    vw_throw(NoImplErr() << "geodiff can't difference DEMs which have differing "
             << "datum radii or meridian offsets.\n");
  }
  if (georef1.datum().semi_major_axis() == georef2.datum().semi_major_axis() &&
      georef1.datum().semi_minor_axis() == georef2.datum().semi_minor_axis() &&
      georef1.datum().meridian_offset() == georef2.datum().meridian_offset() &&
      georef1.datum().proj4_str()       != georef2.datum().proj4_str()) {
    vw_out(WarningMessage) << "Found DEMs with same datum radii and meridian offsets but "
                           << "different projection strings. Use some caution.\n";
  }
  
}

void dem2dem_diff(Options& opt){
  
  DiskImageResourceGDAL dem1_rsrc(opt.dem1_file), dem2_rsrc(opt.dem2_file);
  double dem1_nodata = opt.nodata_value, dem2_nodata = opt.nodata_value;
  if (dem1_rsrc.has_nodata_read()) {
    dem1_nodata = dem1_rsrc.nodata_read();
    opt.nodata_value = dem1_nodata;
    vw_out() << "\tFound input nodata value for DEM 1: " << dem1_nodata << endl;
    vw_out() << "Using this nodata value on output.\n";
  }
  if (dem2_rsrc.has_nodata_read()) {
    dem2_nodata = dem2_rsrc.nodata_read();
    vw_out() << "\tFound input nodata value for DEM 2: " << dem2_nodata << endl;
  }

  if (dem1_rsrc.channels() != 1 || dem2_rsrc.channels() != 1)
      vw_throw(ArgumentErr() << "The input DEMs must have a single channel.\n");
  
  DiskImageView<double> dem1_disk_image_view(dem1_rsrc), dem2_disk_image_view(dem2_rsrc);

  GeoReference dem1_georef, dem2_georef;
  bool has_georef1 = read_georeference(dem1_georef, dem1_rsrc);
  bool has_georef2 = read_georeference(dem2_georef, dem2_rsrc);
  if (!has_georef1 || !has_georef2) 
    vw_throw(ArgumentErr() << "geodiff cannot difference files without a georeference.\n");
  
  georef_sanity_checks(dem1_georef, dem2_georef);

  // Generate a bounding box that is the minimum of the two BBox areas
  BBox2 crop_box = bounding_box(dem1_disk_image_view);

  // Transform the second DEM's bounding box to first DEM's pixels
  GeoTransform gt(dem2_georef, dem1_georef);
  BBox2 box21 = gt.forward_bbox(bounding_box(dem2_disk_image_view));
  crop_box.crop(box21);

  if (crop_box.empty()) 
    vw_throw(ArgumentErr() << "The two DEMs do not have a common area.\n");
    
  ImageViewRef<PixelMask<double>> dem2_trans =
    per_pixel_filter(crop(geo_transform
                          (per_pixel_filter(dem_to_geodetic
                                            (create_mask(dem2_disk_image_view, dem2_nodata),
                                             dem2_georef),
                                            MGeodeticToMAltitude()),
                           dem2_georef, dem1_georef,
                           ValueEdgeExtension<PixelMask<double>>(PixelMask<double>())),
                          crop_box), MaskNaN());
  
  ImageViewRef<double> difference;
  if (opt.use_absolute) {
    difference =
      apply_mask(abs(crop(create_mask(dem1_disk_image_view, dem1_nodata), crop_box) - dem2_trans),
                 opt.nodata_value);
  } else {
    difference =
      apply_mask(crop(create_mask(dem1_disk_image_view, dem1_nodata), crop_box) - dem2_trans,
                 opt.nodata_value);
  }
    
  GeoReference crop_georef = crop(dem1_georef, crop_box);
    
  std::string output_file = opt.output_prefix + "-diff.tif";
  vw_out() << "Writing difference file: " << output_file << "\n";
    
  if (opt.use_float) {
    ImageViewRef<float> difference_float = channel_cast<float>(difference);
    boost::scoped_ptr<DiskImageResourceGDAL>
      rsrc(vw::cartography::build_gdal_rsrc(output_file,
                                            difference_float, opt));
    rsrc->set_nodata_write(opt.nodata_value);
    write_georeference(*rsrc, crop_georef);
    block_write_image(*rsrc, difference_float,
                      TerminalProgressCallback("asp", "\t--> Differencing: "));
  } else {
    boost::scoped_ptr<DiskImageResourceGDAL>
      rsrc(vw::cartography::build_gdal_rsrc(output_file,
                                            difference, opt));
    rsrc->set_nodata_write(opt.nodata_value);
    write_georeference(*rsrc, crop_georef);
    block_write_image(*rsrc, difference,
                      TerminalProgressCallback("asp", "\t--> Differencing: "));
  }
}

// From a DEM, subtract a csv file. Reverse the sign is 'reverse' is true.
void dem2csv_diff(Options & opt, std::string const& dem_file,
                  std::string const & csv_file, bool reverse){
  
  if (opt.csv_format_str == "")
    vw_throw(ArgumentErr() << "CSV files were passed in, but the "
             << "CSV format string was not set.\n");

  // Read the DEM
  DiskImageView<double> dem(dem_file);

  // Read the no-data
  double dem_nodata = opt.nodata_value;
  {
    // Use a scope to free up fast this handle
    DiskImageResourceGDAL dem_rsrc(dem_file);
    if (dem_rsrc.has_nodata_read()) {
      dem_nodata = dem_rsrc.nodata_read();
      opt.nodata_value = dem_nodata;
      vw_out() << "\tFound input nodata value for DEM: " << dem_nodata << endl;
    }
    
    if (dem_rsrc.channels() != 1)
      vw_throw(ArgumentErr() << "The input DEM must have a single channel.\n");
  }
  
  // Read the DEM georef
  GeoReference dem_georef;
  bool has_georef = read_georeference(dem_georef, dem_file);
  if (!has_georef) 
    vw_throw(ArgumentErr() << "geodiff cannot load a georeference from: " << dem_file << ".\n");

  if (opt.csv_proj4_str == "") {
    // Copy from the DEM
    opt.csv_proj4_str = dem_georef.overall_proj4_str();
  }
  
  // Configure a CSV converter object according to the input parameters
  asp::CsvConv csv_conv;
  csv_conv.parse_csv_format(opt.csv_format_str, opt.csv_proj4_str); // Modifies csv_conv
  if (!csv_conv.is_configured()) 
    vw_throw(ArgumentErr() << "Could not configure the csv parser.\n");

  // Set the georef for CSV files
  GeoReference csv_georef = dem_georef;
  csv_conv.parse_georef(csv_georef);

  std::list<asp::CsvConv::CsvRecord> csv_records;
  typedef std::list<asp::CsvConv::CsvRecord>::const_iterator RecordIter;
  csv_conv.read_csv_file(csv_file, csv_records);
  
  std::vector<Vector3> csv_llh;
  for (RecordIter iter = csv_records.begin(); iter != csv_records.end(); iter++) {
    Vector3 xyz = csv_conv.csv_to_cartesian(*iter, csv_georef);
    if (xyz == Vector3() || xyz != xyz)
      continue; // invalid point
    Vector3 llh = dem_georef.datum().cartesian_to_geodetic(xyz); // use the dem's datum
    csv_llh.push_back(llh);
  }

  // We will interpolate into the DEM to find the difference
  ImageViewRef<PixelMask<double>> interp_dem
    = interpolate(create_mask(dem, dem_nodata),
		  BilinearInterpolation(), ConstantEdgeExtension());

  // Save the diffs
  int    count     = 0;
  double diff_min  = std::numeric_limits<double>::max();
  double diff_max  = -diff_min;
  double diff_mean = 0.0;
  double diff_std  = 0.0;

  std::vector<Vector3> csv_diff;
  std::vector<double> csv_errs;
  for (size_t it = 0; it < csv_llh.size(); it++) {

    Vector3 llh = csv_llh[it];
    Vector2 ll  = subvector(llh, 0, 2);
    Vector2 pix = dem_georef.lonlat_to_pixel(ll);
    
    // Check for out of range
    if (pix[0] < 0 || pix[0] > dem.cols() - 1) continue;
    if (pix[1] < 0 || pix[1] > dem.rows() - 1) continue;
    PixelMask<double> dem_ht = interp_dem(pix[0], pix[1]);
    if (!is_valid(dem_ht))
      continue;

    double diff = dem_ht.child() - llh[2];
    if (reverse) 
      diff *= -1;
    if (opt.use_absolute)
      diff = std::abs(diff);

    if (diff > diff_max) diff_max = diff;
    if (diff < diff_min) diff_min = diff;

    diff_mean += diff;
    diff_std  += diff*diff;
    count     += 1;
    csv_diff.push_back(Vector3(ll[0], ll[1], diff));
    csv_errs.push_back(diff);
  }

  if (count > 0) {
    diff_mean /= count;
    diff_std = diff_std/count - diff_mean*diff_mean;
    if (diff_std < 0)
      diff_std = 0; // just in case, for numerical noise
    diff_std = std::sqrt(diff_std);
  }

  double diff_median = 0.0;
  std::sort(csv_errs.begin(), csv_errs.end());
  if (csv_errs.size() > 0) 
    diff_median = csv_errs[csv_errs.size()/2];

  vw_out() << "Max difference:       " << diff_max    << " meters" << std::endl;
  vw_out() << "Min difference:       " << diff_min    << " meters" << std::endl;
  vw_out() << "Mean difference:      " << diff_mean   << " meters" << std::endl;
  vw_out() << "StdDev of difference: " << diff_std    << " meters" << std::endl;
  vw_out() << "Median difference:    " << diff_median << " meters" << std::endl;

  std::string output_file = opt.output_prefix + "-diff.csv";
  vw_out() << "Writing difference file: " << output_file << "\n";
  std::ofstream outfile( output_file.c_str() );
  outfile.precision(16);
  outfile << "# longitude,latitude, height diff (m)" << std::endl;
  outfile << "# " << dem_georef.datum() << std::endl; // dem's datum
  outfile << "# Max difference:       " << diff_max    << " meters" << std::endl;
  outfile << "# Min difference:       " << diff_min    << " meters" << std::endl;
  outfile << "# Mean difference:      " << diff_mean   << " meters" << std::endl;
  outfile << "# StdDev of difference: " << diff_std    << " meters" << std::endl;
  outfile << "# Median difference:    " << diff_median << " meters" << std::endl;
  for (size_t it = 0; it < csv_diff.size(); it++) {
    Vector3 diff = csv_diff[it];
    outfile << diff[0] << "," << diff[1] << "," << diff[2] << std::endl;
  }
}

// Subtract from the first dem the second. One of them can be a CSV file.
int main(int argc, char *argv[]) {

  Options opt;
  try {
    handle_arguments(argc, argv, opt);

    bool is_dem1_csv = asp::is_csv(opt.dem1_file);
    bool is_dem2_csv = asp::is_csv(opt.dem2_file);

    if (is_dem1_csv && is_dem2_csv) 
      vw_throw(ArgumentErr()
               << "Cannot do the diff of two csv files. One of them "
               << "can be converted to a DEM using point2dem fist.\n");
    
    bool reverse = false; // true if first DEM is a csv
    if (is_dem1_csv) {
      reverse = true;
      dem2csv_diff(opt, opt.dem2_file, opt.dem1_file, reverse);
    }else if (is_dem2_csv){
      reverse = false;
      dem2csv_diff(opt, opt.dem1_file, opt.dem2_file, reverse);
    }else{
      // Both are regular DEMs
      dem2dem_diff(opt);
    }

  } ASP_STANDARD_CATCHES;
  
  return 0;
}
