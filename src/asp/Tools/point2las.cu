#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file point2las.cc
///

#include <fstream>
#include <iostream>
#include <string>
#include <boost/program_options.hpp>

// Turn off warnings about things we can't control
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations"
#include <liblas/liblas.hpp>
#pragma GCC diagnostic pop

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
#include <asp/Core/PointUtils.h>

#include <vw/Cartography/PointImageManipulation.h>
#include <vw/Core/Stopwatch.h>
#include <vw/Math/Statistics.h>

using namespace vw;
namespace po = boost::program_options;

// A class to collect some positive errors, and return the error at
// given percentile multiplied by given factor.
class PercentileErrorAccum : public ReturnFixedType<void> {
  typedef double accum_type;
  std::vector<accum_type> m_vals;
public:
  typedef accum_type value_type;
  
  PercentileErrorAccum() { m_vals.clear(); }
  
  void operator()( accum_type const& value ) {
    // Don't add zero errors, those most likely came from invalid points
    if (value > 0)
      m_vals.push_back(value);
  }
  
  int size(){
    return m_vals.size();
  }
  
  value_type value(Vector2 const& outlier_removal_params, bool use_tukey_outlier_removal){

    // Care here with empty sets
    if (m_vals.empty()) {
      vw_out() << "Found no positive triangulation errors in the sample.\n";
      return 0.0;
    }
    
    std::sort(m_vals.begin(), m_vals.end());

    int len = m_vals.size();

    vw_out() << "Collected a sample of " << len << " positive triangulation errors.\n";

    double mean = vw::math::mean(m_vals);
    
    vw_out() << "For this sample: "
             << "min = "     << m_vals.front()
             << ", mean = "  << mean
             << ", stdev = " << vw::math::standard_deviation(m_vals, mean)
             << ", max = " << m_vals.back() << "." << std::endl;

    int i25 = round((len - 1) * 0.25);
    int i50 = round((len - 1) * 0.50);
    int i75 = round((len - 1) * 0.75);

    double Q1 = m_vals[i25];
    double Q2 = m_vals[i50];
    double Q3 = m_vals[i75];
    vw_out() << "Error percentiles: " 
             << "Q1 (25%): " << Q1 << ", "
             << "Q2 (50%): " << Q2 << ", "
             << "Q3 (75%): " << Q3 << "."
             << std::endl;

    if (use_tukey_outlier_removal) {
      vw_out() << "Using as outlier cutoff the Tukey formula Q3 + 1.5*(Q3 - Q1)." << std::endl;
      return Q3 + 1.5*(Q3 - Q1);
    }
    
    double pct    = outlier_removal_params[0]/100.0; // e.g., 0.75
    double factor = outlier_removal_params[1];
    int k         = (int)round((len - 1) * pct);
    
    vw_out() << "Using as outlier cutoff the " << outlier_removal_params[0] << " percentile times "
             << factor << "." << std::endl;
    
    return m_vals[k] * factor;
  }

};

struct Options : vw::GdalWriteOptions {
  // Input
  std::string reference_spheroid, datum;
  std::string pointcloud_file;
  std::string target_srs_string;
  bool        compressed, use_tukey_outlier_removal;
  Vector2     outlier_removal_params;
  double      max_valid_triangulation_error;
  double      triangulation_error_factor;
  int         num_samples;
  
  // Output
  std::string out_prefix;
  Options() : compressed(false), max_valid_triangulation_error(0.0), num_samples(0) {}
};

void handle_arguments( int argc, char *argv[], Options& opt ) {

  po::options_description general_options("General Options");
  general_options.add_options()
    ("compressed,c", po::bool_switch(&opt.compressed)->default_value(false)->implicit_value(true),
     "Compress using laszip.")
    ("output-prefix,o", po::value(&opt.out_prefix), "Specify the output prefix.")
    ("datum", po::value(&opt.datum),
          "Create a geo-referenced LAS file in respect to this datum. Options: WGS_1984, D_MOON (1,737,400 meters), D_MARS (3,396,190 meters), MOLA (3,396,000 meters), NAD83, WGS72, and NAD27. Also accepted: Earth (=WGS_1984), Mars (=D_MARS), Moon (=D_MOON).")
    ("reference-spheroid,r", po::value(&opt.reference_spheroid),
     "This is identical to the datum option.")
    ("t_srs", po::value(&opt.target_srs_string)->default_value(""),
     "Specify a custom projection (PROJ.4 string).")
    ("remove-outliers-params", po::value(&opt.outlier_removal_params)->default_value(Vector2(75.0, 3.0), "pct factor"),
     "Outlier removal based on percentage. Points with triangulation error larger than pct-th percentile times factor will be removed as outliers. [default: pct=75.0, factor=3.0]")
    ("use-tukey-outlier-removal", po::bool_switch(&opt.use_tukey_outlier_removal)->default_value(false)->implicit_value(true),
     "Remove outliers above Q3 + 1.5*(Q3 - Q1). Takes precedence over the above approach.")

    ("max-valid-triangulation-error", po::value(&opt.max_valid_triangulation_error)->default_value(0.0),
     "Outlier removal based on threshold. Points with triangulation error larger than this, if positive (measured in meters) will be removed from the cloud. Takes precedence over the above methods.")
    ("triangulation-error-factor", po::value(&opt.triangulation_error_factor)->default_value(0.0),
     "If this factor is positive, save the point cloud triangulation error to the 2-byte LAS intensity field by storing min(round(factor*error), 65535). Resulting values that equal 65535 should be treated with caution.")
    ("num-samples-for-outlier-estimation", po::value(&opt.num_samples)->default_value(1000000),
     "Approximate number of samples to pick from the input cloud to find the outlier cutoff based on triangulation error.");
  
  general_options.add( vw::GdalWriteOptionsDescription(opt) );

  po::options_description positional("");
  positional.add_options()
    ("input-file", po::value(&opt.pointcloud_file), "Input Point Cloud");

  po::positional_options_description positional_desc;
  positional_desc.add("input-file", 1);

  std::string usage("[options] <point-cloud>");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line( argc, argv, opt, general_options, general_options,
                             positional, positional_desc, usage,
                             allow_unregistered, unregistered );

  if ( opt.pointcloud_file.empty() )
    vw_throw( ArgumentErr() << "Missing point cloud.\n"
              << usage << general_options );

  if ( opt.out_prefix.empty() )
    opt.out_prefix =
      vw::prefix_from_filename( opt.pointcloud_file );

  // reference_spheroid and datum are aliases.
  boost::to_lower(opt.reference_spheroid);
  boost::to_lower(opt.datum);
  if (opt.datum != "" && opt.reference_spheroid != "")
    vw_throw( ArgumentErr() << "Both --datum and --reference-spheroid were specified.\n");
  if (opt.datum == "")
    opt.datum = opt.reference_spheroid;

  double pct = opt.outlier_removal_params[0], factor = opt.outlier_removal_params[1];
  if (pct <= 0.0 || pct > 100.0 || factor <= 0.0){
    vw_throw( ArgumentErr() << "Invalid values were provided for outlier removal parameters.\n");
  }

  if (opt.max_valid_triangulation_error < 0.0) 
    vw_throw( ArgumentErr() << "The maximum valid triangulation error must be non-negative.\n");

  if (opt.num_samples <= 0) 
    vw_throw( ArgumentErr() << "Must pick a positive number of samples.\n");

  // Create the output directory
  vw::create_out_dir(opt.out_prefix);

  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);
}

void find_error_image_and_do_stats(Options& opt, ImageViewRef<double> & error_image) {
      
  std::vector<std::string> pointcloud_files;
  pointcloud_files.push_back(opt.pointcloud_file);
  error_image = asp::point_cloud_error_image(pointcloud_files);
  
  if (error_image.rows() == 0 || error_image.cols() == 0) {
    vw_out() << "The point cloud files must have an equal number of channels which "
             << "must be 4 or 6 to be able to remove outliers.\n";
    opt.max_valid_triangulation_error = 0.0;
    return;
  }

  if (opt.max_valid_triangulation_error > 0.0) {
    vw_out() << "Using the set maximum valid triangulation error as outlier cutoff: "
             << opt.max_valid_triangulation_error << "." << std::endl;
    return;
  }
    
  vw_out() << "Estimating the maximum valid triangulation error (outlier cutoff).\n";
    
  int num_err_cols = error_image.cols();
  int num_err_rows = error_image.rows();
    
  double area = std::max(num_err_cols * num_err_rows, 1);
  int sample_rate = round(sqrt(double(area) / double(opt.num_samples)));
  if (sample_rate < 1) 
    sample_rate = 1;
    
  Stopwatch sw;
  sw.start();
  PixelAccumulator<PercentileErrorAccum> error_accum;
  for_each_pixel(subsample(error_image, sample_rate),
                 error_accum,
                 TerminalProgressCallback
                 ("asp","Error estim : ") );

  opt.max_valid_triangulation_error = error_accum.value(opt.outlier_removal_params,
                                                        opt.use_tukey_outlier_removal);
  
  sw.stop();
  vw_out(DebugMessage, "asp") << "Elapsed time: " << sw.elapsed_seconds() << std::endl;
  vw_out() << "Found the maximum valid triangulation error (outlier cutoff): "
           << opt.max_valid_triangulation_error << "." << std::endl;
}

int main( int argc, char *argv[] ) {
  
  // TODO(oalexan1): need to understand what is the optimal strategy
  // for traversing the input point cloud file to minimize the reading
  // time.

  Options opt;
  try {
    handle_arguments(argc, argv, opt);

    ImageViewRef<double> error_image;
    if (opt.outlier_removal_params[0] < 100.0 || opt.max_valid_triangulation_error > 0.0)
      find_error_image_and_do_stats(opt, error_image);

    // Save the las file in respect to a reference spheroid if provided
    // by the user.
    liblas::Header header;
    cartography::Datum datum;
    bool have_user_datum = asp::read_user_datum(0, 0, opt.datum, datum);

    cartography::GeoReference georef;
    bool have_input_georef = vw::cartography::read_georeference(georef, opt.pointcloud_file);
    if (have_input_georef && opt.target_srs_string.empty()) {
      opt.target_srs_string = georef.overall_proj4_str();
    }

    bool is_geodetic = false;
    if (have_user_datum || !opt.target_srs_string.empty()){

      // Set the srs string into georef.
      asp::set_srs_string(opt.target_srs_string,
                          have_user_datum, datum,
                          have_input_georef, georef);
      liblas::SpatialReference ref;
      std::string target_srs = georef.overall_proj4_str();

      ref.SetFromUserInput(target_srs);
      vw_out() << "Using projection string: '" << target_srs << "'"<< std::endl;
      header.SetSRS(ref);

      is_geodetic = true;
      datum = georef.datum();
    }

    // Save the las file with given georeference, if present
    ImageViewRef<Vector3> point_image = asp::read_asp_point_cloud<3>(opt.pointcloud_file);
    if (is_geodetic) {
      point_image = cartesian_to_geodetic(point_image, datum);
      double avg_lon = asp::find_avg_lon(point_image); // see if to use [-180, 180] or [0, 360]
      point_image = geodetic_to_point(asp::recenter_longitude(point_image, avg_lon), georef);
    }

    BBox3 cloud_bbox = asp::pointcloud_bbox(point_image, is_geodetic);

    // The las format stores the values as 32 bit integers. So, for a
    // given point, we store round((point-offset)/scale), as well as
    // the offset and scale values. Here we decide the values for
    // offset and scale to lose minimum amount of precision. We make
    // the scale almost as large as it can be without causing integer overflow.
    Vector3 offset = (cloud_bbox.min() + cloud_bbox.max())/2.0;
    double  maxInt = std::numeric_limits<int32>::max();
            maxInt *= 0.95; // Just in case stay a bit away
    Vector3 scale  = cloud_bbox.size()/(2.0*maxInt);
    for (size_t i = 0; i < scale.size(); i++){
      if (scale[i] <= 0.0) scale[i] = 1.0e-16; // avoid degeneracy
    }

    // The line below causes trouble with compression in libLAS-1.7.0.
    //header.SetDataFormatId(liblas::ePointFormat1);
    header.SetScale (scale [0], scale [1], scale [2]);
    header.SetOffset(offset[0], offset[1], offset[2]);

    // Populate the min and max fields of the LAS header
    header.SetMax(cloud_bbox.max().x(),cloud_bbox.max().y(),cloud_bbox.max().z());
    header.SetMin(cloud_bbox.min().x(),cloud_bbox.min().y(),cloud_bbox.min().z());

    std::string lasFile;
    header.SetCompressed(opt.compressed);
    if (opt.compressed)
      lasFile = opt.out_prefix + ".laz";
    else
      lasFile = opt.out_prefix + ".las";

    vw_out() << "Writing LAS file: " << lasFile + "\n";
    std::ofstream ofs;
    ofs.open(lasFile.c_str(), std::ios::out | std::ios::binary);
    liblas::Writer writer(ofs, header);

    TerminalProgressCallback tpc("asp", "\t--> ");
    long long int num_total_points = 0;
    long long int num_kept_points = 0;
    
    for (int row = 0; row < point_image.rows(); row++){
      tpc.report_fractional_progress(row, point_image.rows());
      for (int col = 0; col < point_image.cols(); col++){

        Vector3 point = point_image(col, row);

        // Skip no-data points
        bool is_good = ( (!is_geodetic && point != vw::Vector3()) ||
                         (is_geodetic  && !boost::math::isnan(point.z())) );
        if (!is_good) continue;

        num_total_points++;
        
        if (opt.max_valid_triangulation_error > 0.0 &&
            error_image(col, row) > opt.max_valid_triangulation_error) 
          continue;

        num_kept_points++;
#if 0
        // For comparison later with las2txt.
        std::cout.precision(16);
        std::cout << "\npoint " << point[0] << ' ' << point[1] << ' '
                  << point[2] << std::endl;
#endif

        liblas::Point las_point(&header);
        las_point.SetCoordinates(point[0], point[1], point[2]);

        if (opt.triangulation_error_factor > 0.0) {
          // Scale the triangulation error, clamp it, and save it as
          // uint16.  The LAS 1.2 format has no fields (apart from the
          // taken already x, y, and z) with 32-bit values, so uint16
          // is all one can do.
          double scaled_error = opt.triangulation_error_factor * error_image(col, row);
          scaled_error = round(scaled_error);
          scaled_error = std::max(scaled_error, 0.0); // should not be necessary
          scaled_error = std::min(scaled_error, double(std::numeric_limits<std::uint16_t>::max()));
          las_point.SetIntensity(std::uint16_t(scaled_error));
        }
          
        writer.WritePoint(las_point);
      }
    }
    tpc.report_finished();

    vw_out () << "Saved: " << num_kept_points << " points." << std::endl;
    
    if (opt.max_valid_triangulation_error > 0.0) {
      long long int num_excluded = num_total_points - num_kept_points;
      double percent = 100.0 * double(num_excluded)/num_total_points;
      percent = round(percent * 100.0)/100.0; // don't keep too many digits
      vw_out() << "Excluded based on triangulation error " << num_excluded << " points ("
               << percent << "%)." << std::endl;
    }
    
  } ASP_STANDARD_CATCHES;

  return 0;
}
