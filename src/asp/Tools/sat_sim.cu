#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// Tool to create simulated satellite images and/or pinhole cameras for them.
// See the manual for details.

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
#include <asp/Camera/SyntheticLinescan.h>
#include <asp/Camera/SatSim.h>

#include <vw/Camera/PinholeModel.h>
#include <vw/Core/StringUtils.h>
#include <vw/Core/Stopwatch.h>

namespace po = boost::program_options;
namespace fs = boost::filesystem;

void handle_arguments(int argc, char *argv[], asp::SatSimOptions& opt) {

  double NaN = std::numeric_limits<double>::quiet_NaN();
  po::options_description general_options("General options");
  general_options.add_options()
    ("dem", po::value(&opt.dem_file)->default_value(""), "Input DEM file.")
    ("ortho", po::value(&opt.ortho_file)->default_value(""), "Input georeferenced image file.")
    ("output-prefix,o", po::value(&opt.out_prefix), "Specify the output prefix. All the "
    "files that are saved will start with this prefix.")
    ("camera-list", po::value(&opt.camera_list)->default_value(""),
     "A file containing the list of pinhole cameras to create synthetic images for. "
     "Then these cameras will be used instead of generating them. Specify one file "
     "per line. The options --first, --last, --num, --focal-length, "
     "and --optical-center will be ignored.")
    ("first", po::value(&opt.first)->default_value(vw::Vector3(), ""),
    "First camera position, specified as DEM pixel column and row, and height above "
    "the DEM datum.")
    ("last", po::value(&opt.last)->default_value(vw::Vector3(), ""),
    "Last camera position, specified as DEM pixel column and row, and height above "
    "the DEM datum.")
    ("num", po::value(&opt.num_cameras)->default_value(0),
    "Number of cameras to generate, including the first and last ones. Must be positive. "
    "The cameras are uniformly distributed along the straight edge from first to last (in "
    "projected coordinates).")
    ("first-ground-pos", po::value(&opt.first_ground_pos)->default_value(vw::Vector2(NaN, NaN), ""),
    "Coordinates of first camera ground footprint center (DEM column and row). "
    "If not set, the cameras will look straight down (perpendicular to along "
    "and across track directions).")
    ("last-ground-pos", po::value(&opt.last_ground_pos)->default_value(vw::Vector2(NaN, NaN), ""),
    "Coordinates of last camera ground footprint center (DEM column and row). "
    "If not set, the cameras will look straight down (perpendicular to along "
    "and across track directions).")
    ("focal-length", po::value(&opt.focal_length)->default_value(NaN),
     "Output camera focal length in units of pixel.")
    ("optical-center", po::value(&opt.optical_center)->default_value(vw::Vector2(NaN, NaN),"NaN NaN"),
     "Output camera optical center (image column and row). Units of pixel.")
    ("image-size", po::value(&opt.image_size)->default_value(vw::Vector2(NaN, NaN),
      "NaN NaN"),
      "Output camera image size (width and height).")
    ("roll", po::value(&opt.roll)->default_value(NaN),
    "Camera roll angle, in degrees. See the documentation for more details.")
    ("pitch", po::value(&opt.pitch)->default_value(NaN),
     "Camera pitch angle, in degrees.")
    ("yaw", po::value(&opt.yaw)->default_value(NaN),
     "Camera yaw angle, in degrees.")
    ("velocity", po::value(&opt.velocity)->default_value(NaN),
     "Satellite velocity, in meters per second. Used for modeling jitter. A value of "
     "around 8000 m/s is typical for a satellite like SkySat in Sun-synchronous orbit "
     "(90 minute period) at an altitude of about 450 km. For WorldView, the velocity "
     "is around 7500 m/s, with a higher altitude and longer period.")
    ("jitter-frequency", po::value(&opt.jitter_frequency_str)->default_value(""),
     "Jitter frequency, in Hz. Used for modeling jitter (satellite vibration). "
     "Several frequencies can be specified. Use a quoted list, with spaces or "
     "commas as separators. See also  --jitter-amplitude and --horizontal-uncertainty.")
    ("jitter-phase", po::value(&opt.jitter_phase_str)->default_value(""),
     "Jitter phase, in radians. Measures the jitter phase offset from the start of "
     "the orbit as set by ``--first``. Specify as a quoted list of numbers. Number "
     "of values must be 3 times the number of frequencies. The order in this list "
     "corresponds to phase for roll, pitch, and yaw for first frequency, then "
     "second frequency, etc. If not specified, will be set to 0.")
    ("horizontal-uncertainty", po::value(&opt.horizontal_uncertainty_str)->default_value(""),
     "Camera horizontal uncertainty on the ground, in meters, at nadir orientation. "
     "Specify as a quoted list of three numbers, used for roll, pitch, and yaw. The "
     "jitter amplitude for each of these angles is found as "
     "= atan(horizontal_uncertainty / satellite_elevation_above_datum), then converted "
     "to degrees. See also --jitter-amplitude.")
    ("jitter-amplitude", po::value(&opt.jitter_amplitude_str)->default_value(""),
     "Jitter amplitude, in micro radians. Specify as a quoted list having "
     "amplitude in roll, pitch, yaw for first frequency, then for second, and so on. "
     "Separate the values by spaces or commas.")
    ("first-index", po::value(&opt.first_index)->default_value(-1),
     "Index of first camera and/or image to generate, starting from 0. If not set, will create "
     "all images/cameras. This is used for parallelization.")
    ("last-index", po::value(&opt.last_index)->default_value(-1),
     "Index of last image and/or camera to generate, starting from 0. Stop before this index. "
     "If not set, will create all images/cameras. This is used for parallelization.")
    ("frame-rate", po::value(&opt.frame_rate)->default_value(NaN), 
     "Camera frame rate, per second. Can be in double precision. If set, it will override "
     "--num. The cameras will start from --first (after any position adjustment, if "
     "applicable, per the doc). Set the --velocity value. The last camera will be no further "
     "than the (adjusted) value of --last along the orbit.")
     ("sensor-type", po::value(&opt.sensor_type)->default_value("pinhole"),
      "Sensor type for created cameras and images. Can be one of: pinhole, linescan.")
    ("square-pixels", po::bool_switch(&opt.square_pixels)->default_value(false)->implicit_value(true),
      "When creating linescan images, override the image height (the second value "
      "in --image-size) to ensure that the horizontal and vertical ground "
      "sample distances are very similar.")
    ("no-images", po::bool_switch(&opt.no_images)->default_value(false)->implicit_value(true),
     "Create only cameras, and no images. Cannot be used with --camera-list.")
     ("save-ref-cams", po::bool_switch(&opt.save_ref_cams)->default_value(false)->implicit_value(true),
     "For each created camera, save also the 'reference' camera that has no roll, pitch, "
     "yaw, jitter, or 90 degree in-sensor-plane rotation from camera to satellite " 
     "coordinates. Their names have '-ref-' after the output prefix.")
    ("save-as-csm", 
      po::bool_switch(&opt.save_as_csm)->default_value(false)->implicit_value(true),
      "Save Pinhole (frame) cameras in the CSM format, as done for linescan cameras. "
      "Can be used to combine these sensors in bundle adjustment and solving for jitter")
    ("dem-height-error-tol", po::value(&opt.dem_height_error_tol)->default_value(0.001),
     "When intersecting a ray with a DEM, use this as the height error tolerance "
     "(measured in meters). It is expected that the default will be always good enough.")
    ;
  general_options.add(vw::GdalWriteOptionsDescription(opt));

  po::options_description positional("");
  po::positional_options_description positional_desc;

  std::string usage("--dem <dem file> --ortho <ortho image file> "
                    "[other options]");

  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                            positional, positional_desc, usage,
                            allow_unregistered, unregistered);

  if (opt.dem_file == "" || opt.ortho_file == "")
    vw::vw_throw(vw::ArgumentErr() << "Missing input DEM and/or ortho image.\n");
  if (opt.out_prefix == "")
    vw::vw_throw(vw::ArgumentErr() << "Missing output prefix.\n");

  if (std::isnan(opt.image_size[0]) || std::isnan(opt.image_size[1]))
    vw::vw_throw(vw::ArgumentErr() << "The image size must be specified.\n");
  if (opt.image_size[0] <= 1 || opt.image_size[1] <= 1)
    vw::vw_throw(vw::ArgumentErr() << "The image size must be at least 2 x 2.\n");

  if (opt.camera_list != "" && opt.no_images)
    vw::vw_throw(vw::ArgumentErr() << "The --camera-list and --no-images options "
      "cannot be used together.\n");
  
  if (opt.camera_list != "" && opt.square_pixels)
    vw::vw_throw(vw::ArgumentErr() << "The --camera-list and --square-pixels options "
      "cannot be used together. Making the mapprojected pixels be approximately "
      "square requires modifying the camera model.\n");

  if (opt.camera_list == "") {
    if (opt.first == vw::Vector3() || opt.last == vw::Vector3())
      vw::vw_throw(vw::ArgumentErr() << "The first and last camera positions must be "
        "specified.\n");

    if (opt.first[2] != opt.last[2])
      vw::vw_out() << "Warning: The first and last camera positions have different "
                   << "heights above the datum. This is supported but is not usual. "
                   << "Check your inputs.\n";
    
    if (std::isnan(opt.frame_rate)) {
      if (opt.num_cameras < 1)
        vw::vw_throw(vw::ArgumentErr() << "The number of cameras must be at least 1.\n");
      if (opt.num_cameras == 1)
        vw::vw_out(vw::WarningMessage) << "Warning: Creating only one camera.\n";
    } else {
      // Frame rate is set. Then need not set num cameras.
      if (opt.num_cameras > 0)
        vw::vw_throw(vw::ArgumentErr() << "Cannot set both --num and --frame-rate.\n");
      // Must have a positive velocity
      if (std::isnan(opt.velocity) || opt.velocity <= 0.0)
        vw::vw_throw(vw::ArgumentErr() << "The velocity must be positive if using "
         << " --frame-rate.\n");
    }

    // Validate focal length, optical center, and image size
    if (std::isnan(opt.focal_length))
      vw::vw_throw(vw::ArgumentErr() << "The focal length must be positive.\n");
    if (std::isnan(opt.optical_center[0]) || std::isnan(opt.optical_center[1]))
      vw::vw_throw(vw::ArgumentErr() << "The optical center must be specified.\n");

    // Either both first and last ground positions are specified, or none.
    if (std::isnan(norm_2(opt.first_ground_pos)) != 
      std::isnan(norm_2(opt.last_ground_pos)))
      vw::vw_throw(vw::ArgumentErr() << "Either both first and last ground positions "
        "must be specified, or none.\n");

    // Check that either all of roll, pitch, and yaw are specified, or none.
    int ans = int(std::isnan(opt.roll)) +
              int(std::isnan(opt.pitch)) +
              int(std::isnan(opt.yaw));
    if (ans != 0 && ans != 3)
      vw::vw_throw(vw::ArgumentErr() << "Either all of roll, pitch, and yaw must be "
        "specified, or none.\n");
  }

  // Parse jitter frequency
  // Convert from string to vector of doubles
  std::string sep = ", \t\n"; // separators: comma, space, tab, newline
  opt.jitter_frequency = vw::str_to_std_vec(opt.jitter_frequency_str, sep);
  if (opt.jitter_frequency.empty())
    opt.jitter_frequency.push_back(NaN);

  // Horizontal uncertainty must be 3 values. Must specify either this or 
  // jitter amplitude.
  opt.horizontal_uncertainty 
    = vw::str_to_std_vec(opt.horizontal_uncertainty_str, sep);
  if (!opt.horizontal_uncertainty.empty() && opt.horizontal_uncertainty.size() != 3)
    vw::vw_throw(vw::ArgumentErr() << "The horizontal uncertainty must be specified "
      "as three values, separated by commas or spaces.\n");

  // Number of jitter amplitudes must be 3x the number of frequencies. It can 
  // be empty, if horizontal uncertainty is specified.
  opt.jitter_amplitude = vw::str_to_std_vec(opt.jitter_amplitude_str, sep);
  if (opt.jitter_amplitude.empty() && opt.horizontal_uncertainty.empty()) {
    // Default amplitude is 0
    for (size_t i = 0; i < opt.jitter_frequency.size() * 3; i++)
      opt.jitter_amplitude.push_back(0.0);
  }

  // Number of jitter phases must be 3x the number of frequencies.
  opt.jitter_phase = vw::str_to_std_vec(opt.jitter_phase_str, sep);
  if (opt.jitter_phase.empty()) {
    // Default phase is 0
    for (size_t i = 0; i < opt.jitter_frequency.size() * 3; i++)
      opt.jitter_phase.push_back(0.0);
  }

  // Sanity checks
  if (!opt.horizontal_uncertainty_str.empty() && !opt.jitter_amplitude_str.empty()) 
    vw::vw_throw(vw::ArgumentErr() 
      << "Cannot specify both jitter uncertainty and jitter amplitude.\n");
  if (!opt.horizontal_uncertainty.empty() && !opt.jitter_amplitude.empty()) 
    vw::vw_throw(vw::ArgumentErr() 
      << "Cannot specify both jitter uncertainty and jitter amplitude.\n");

  bool model_jitter = (!std::isnan(opt.jitter_frequency[0]));
  if (model_jitter) {

    bool have_roll_pitch_yaw = !std::isnan(opt.roll) && !std::isnan(opt.pitch) &&
      !std::isnan(opt.yaw);
    if (!have_roll_pitch_yaw)
      vw::vw_throw(vw::ArgumentErr() << "Modelling jitter requires specifying --roll, --pitch, and --yaw.\n");
    
    if (opt.camera_list != "") 
      vw::vw_throw(vw::ArgumentErr() << "The --camera-list option must not be set "
        << "when modeling jitter.\n");

    // See if the user specified either horizontal uncertainty or jitter amplitude
    if (opt.horizontal_uncertainty_str.empty() && opt.jitter_amplitude_str.empty()) 
      vw::vw_throw(vw::ArgumentErr() << "Must specify either horizontal uncertainty "
        << "or jitter amplitude.\n");

    if (3 * opt.jitter_frequency.size() != opt.jitter_phase.size())
      vw::vw_throw(vw::ArgumentErr() << "The number of jitter phases must be "
        << "three times the number of jitter frequencies.\n");

    if (opt.horizontal_uncertainty.empty()) {
      // Jitter amplitude was specified
      if (3 * opt.jitter_frequency.size() != opt.jitter_amplitude.size())
        vw::vw_throw(vw::ArgumentErr() << "The number of jitter amplitudes must be "
          << "three times the number of jitter frequencies.\n");
    } else {
      // Horizontal uncertainty was specified.
      if (opt.horizontal_uncertainty.size() != 3)
        vw::vw_throw(vw::ArgumentErr() << "The number of horizontal uncertainty values "
          << "must be 3.\n");
      
      if (opt.horizontal_uncertainty[0] < 0 || opt.horizontal_uncertainty[1] < 0 ||
          opt.horizontal_uncertainty[2] < 0)
        vw::vw_throw(vw::ArgumentErr() << "The horizontal uncertainty must be non-negative.\n");
    }

    // Check that all jitter frequencies are not NaN and positive
    for (size_t i = 0; i < opt.jitter_frequency.size(); i++) {
      if (std::isnan(opt.jitter_frequency[i]))
        vw::vw_throw(vw::ArgumentErr() << "The jitter frequency must be specified.\n");
      if (opt.jitter_frequency[i] <= 0)
        vw::vw_throw(vw::ArgumentErr() << "The jitter frequency must be positive.\n");
    }
    
  } // end if model jitter

  if (opt.velocity <= 0)
    vw::vw_throw(vw::ArgumentErr() << "The satellite velocity must be positive.\n");

  // Checks for linescan cameras
  if (opt.sensor_type == "linescan" && std::isnan(opt.velocity))
    vw::vw_throw(vw::ArgumentErr() << "The satellite velocity must be specified "
      << "in order to create linescan cameras.\n");
  if (opt.square_pixels && opt.sensor_type != "linescan")
    vw::vw_throw(vw::ArgumentErr() << "Cannot specify --square-pixels unless creating "
      << "linescan cameras.\n");

  if (opt.sensor_type == "linescan" && opt.save_as_csm) 
    vw::vw_out(vw::WarningMessage) << "The --save-as-csm option is ignored "
      << "for linescan cameras since there is no other approach in this case.\n";

  // Sanity check the first and last indices
  int ans = int(opt.first_index < 0) + int(opt.last_index < 0);
  if (ans != 0 && ans != 2)
    vw::vw_throw(vw::ArgumentErr() << "Either both first and last indices must be "
      "specified, or none.\n");
  if (ans == 0 && opt.first_index >= opt.last_index)
    vw::vw_throw(vw::ArgumentErr() << "The first index must be less than "
      "the last index.\n");

  // Check for sensor type
  if (opt.sensor_type != "pinhole" && opt.sensor_type != "linescan")
    vw::vw_throw(vw::ArgumentErr() << "The sensor type must be either pinhole or linescan.\n");

  // Create the output directory based on the output prefix
  vw::create_out_dir(opt.out_prefix);

  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);

  return;
}

int main(int argc, char *argv[]) {

  asp::SatSimOptions opt;
  try {
    handle_arguments(argc, argv, opt);

    // Read the DEM
    vw::ImageViewRef<vw::PixelMask<float>> dem;
    float dem_nodata_val = -std::numeric_limits<float>::max(); // will change
    vw::cartography::GeoReference dem_georef;
    asp::readGeorefImage(opt.dem_file, dem_nodata_val, dem_georef, dem);
    double height_guess = asp::findDemHeightGuess(dem); // useful for ray-dem intersection

    // Read the ortho image
    vw::ImageViewRef<vw::PixelMask<float>> ortho;
    float ortho_nodata_val = -std::numeric_limits<float>::max(); // will change
    vw::cartography::GeoReference ortho_georef;
    asp::readGeorefImage(opt.ortho_file, ortho_nodata_val, ortho_georef, ortho);

    std::vector<std::string> cam_names;
    std::vector<vw::CamPtr> cams;
    // smart point
    bool external_cameras = false;
    if (!opt.camera_list.empty()) {
      // Read the cameras
      if (opt.sensor_type == "pinhole")
        asp::readPinholeCameras(opt, cam_names, cams);
      else
        asp::readLinescanCameras(opt, cam_names, cams);
      external_cameras = true;
    } else {
      // Generate the cameras   
      double orbit_len = 0.0;
      std::map<int, vw::Vector3> trajectory;
      // vector of rot matrices. The matrix cam2world_no_jitter
      // is only needed with linescan cameras, but compute it for consistency 
      // in all cases.
      std::map<int, vw::Matrix3x3> cam2world, cam2world_no_jitter, ref_cam2world;
      asp::calcTrajectory(opt, dem_georef, dem, height_guess,
        orbit_len, trajectory, cam2world, cam2world_no_jitter, ref_cam2world); // outputs
      // Generate cameras
      if (opt.sensor_type == "pinhole")
        asp::genPinholeCameras(opt, dem_georef, trajectory, cam2world, ref_cam2world,
          cam_names, cams);
      else
        asp::genLinescanCameras(orbit_len, dem_georef, dem, trajectory, 
          cam2world, cam2world_no_jitter, ref_cam2world, height_guess,
          opt, cam_names, cams); // outputs
    }

    // Generate images
    if (!opt.no_images)
      asp::genImages(opt, external_cameras, cam_names, cams, dem_georef, dem, 
        height_guess, ortho_georef, ortho, ortho_nodata_val);

  } ASP_STANDARD_CATCHES;

  return 0;
}
