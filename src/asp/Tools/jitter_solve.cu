#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file jitter_adjust.cc
///
/// Use n adjustments for every camera, placed at several lines in the image
// with interpolation between them. The pdf doc has more info.

// TODO(oalexan1): Move some UsgsAstroLsSensorModel functions from
// here and from LinescanDGModel.cc to CsmUtils.cc.

// TODO(oalexan1): Add two passes and outlier filtering. For now
// try to use clean matches.

#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/Sessions/CameraUtils.h>
#include <asp/Camera/CsmModel.h>
#include <asp/Camera/CsmUtils.h>
#include <asp/Camera/BundleAdjustCamera.h>
#include <asp/Camera/JitterSolveCostFuns.h>
#include <asp/Camera/JitterSolveUtils.h>
#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Core/BundleAdjustUtils.h>
#include <asp/Core/IpMatchingAlgs.h> // Lightweight header for matching algorithms
#include <asp/Core/SatSimBase.h>
#include <asp/Core/CameraTransforms.h>

#include <vw/BundleAdjustment/ControlNetwork.h>
#include <vw/BundleAdjustment/ControlNetworkLoader.h>
#include <vw/Core/Stopwatch.h>
#include <vw/Cartography/CameraBBox.h>

#include <usgscsm/UsgsAstroLsSensorModel.h>
#include <usgscsm/UsgsAstroFrameSensorModel.h>
#include <usgscsm/Utilities.h>

#include <xercesc/util/PlatformUtils.hpp>

#include <ceres/ceres.h>
#include <ceres/loss_function.h>

namespace po = boost::program_options;
namespace fs = boost::filesystem;

using namespace vw;
using namespace vw::math;

namespace asp {

struct Options: public asp::BaBaseOptions {
  int num_lines_per_position, num_lines_per_orientation, num_anchor_points;
  double quat_norm_weight, anchor_weight, roll_weight, yaw_weight;
  std::string anchor_dem;
  int num_anchor_points_extra_lines;
  bool initial_camera_constraint;
  std::map<int, int> orbital_groups;
  double forced_triangulation_distance;
};
    
void handle_arguments(int argc, char *argv[], Options& opt) {

  po::options_description general_options("");
  general_options.add_options()
    ("output-prefix,o",  po::value(&opt.out_prefix), "Prefix for output filenames.")
    ("session-type,t",   po::value(&opt.stereo_session)->default_value(""),
     "Select the stereo session type to use for processing. Usually the program "
     "can select this automatically by the file extension, except for xml cameras. "
     "See the doc for options.")
    ("input-adjustments-prefix",  po::value(&opt.input_prefix),
     "Prefix to read initial adjustments from, written by bundle_adjust. "
     "Not required. Cameras in .json files in ISD or model state format "
     "can be passed in with no adjustments.")
    ("num-lines-per-position", po::value(&opt.num_lines_per_position)->default_value(-1),
     "Resample the input camera positions and velocities, using this many lines per "
     "produced position and velocity. If not set, use the positions and velocities "
     "from the CSM file as they are.")
    ("num-lines-per-orientation", po::value(&opt.num_lines_per_orientation)->default_value(-1),
     "Resample the input camera orientations, using this many lines per produced orientation. "
     "If not set, use the orientations from the CSM file as they are.")
    ("match-first-to-last",
     po::value(&opt.match_first_to_last)->default_value(false)->implicit_value(true),
     "Match first several images to last several images by extending the logic of "
     "--overlap-limit past the last image to the earliest ones.")
    ("overlap-limit",        po::value(&opt.overlap_limit)->default_value(0),
     "Limit the number of subsequent images to search for matches to the current image "
     "to this value. By default match all images.")
    ("match-files-prefix",  po::value(&opt.match_files_prefix)->default_value(""),
     "Use the match files from this prefix instead of the current output prefix.")
    ("clean-match-files-prefix",  po::value(&opt.clean_match_files_prefix)->default_value(""),
     "Use as input match files the *-clean.match files from this prefix.")
    ("min-matches", po::value(&opt.min_matches)->default_value(30),
     "Set the minimum  number of matches between images that will be considered.")
    ("max-pairwise-matches", po::value(&opt.max_pairwise_matches)->default_value(10000),
     "Reduce the number of matches per pair of images to at most this "
     "number, by selecting a random subset, if needed. This happens "
     "when setting up the optimization, and before outlier filtering.")
    ("min-triangulation-angle", po::value(&opt.min_triangulation_angle)->default_value(0.1),
     "The minimum angle, in degrees, at which rays must meet at a triangulated point to "
     "accept this point as valid. It must be a positive value.")
    ("max-initial-reprojection-error", po::value(&opt.max_init_reproj_error)->default_value(10),
     "Filter as outliers triangulated points project using initial cameras with error more than "
     "this, measured in pixels. Since jitter corrections are supposed to be small and cameras "
     "bundle-adjusted by now, this value need not be too big.")
    ("robust-threshold", po::value(&opt.robust_threshold)->default_value(0.5),
     "Set the threshold for the Cauchy robust cost function. Increasing this makes "
     "the solver focus harder on the larger errors.")
    ("parameter-tolerance",  po::value(&opt.parameter_tolerance)->default_value(1e-12),
     "Stop when the relative error in the variables being optimized is less than this.")
    ("num-iterations",       po::value(&opt.num_iterations)->default_value(500),
     "Set the maximum number of iterations.")
    ("tri-weight", po::value(&opt.tri_weight)->default_value(0.0),
     "The weight to give to the constraint that optimized triangulated "
     "points stay close to original triangulated points. A positive "
     "value will help ensure the cameras do not move too far, but a "
     "large value may prevent convergence. Does not apply to GCP or "
     "points constrained by a DEM. This adds a robust cost function  "
     "with the threshold given by --tri-robust-threshold. "
     "The suggested value is 0.1 to 0.5 divided by the image ground "
     "sample distance.")
    ("tri-robust-threshold",
     po::value(&opt.tri_robust_threshold)->default_value(0.1),
     "Use this robust threshold to attenuate large differences "
     "between initial and optimized triangulation points, after multiplying "
     "them by --tri-weight.")
    ("heights-from-dem",   po::value(&opt.heights_from_dem)->default_value(""),
     "If the cameras have already been bundle-adjusted and aligned "
     "to a known DEM, in the triangulated points obtained from "
     "interest point matches replace the heights with the ones from this "
     "DEM before optimizing them while tying the points to this DEM via "
     "--heights-from-dem-weight and --heights-from-dem-robust-threshold.")
    ("heights-from-dem-weight", po::value(&opt.heights_from_dem_weight)->default_value(0.5),
     "How much weight to give to keep the triangulated points close "
     "to the DEM if specified via --heights-from-dem. This value "
     "should be about 0.1 to 0.5 divided by the image ground sample "
     "distance, as then it will convert the measurements from meters to "
     "pixels, which is consistent with the pixel reprojection error term.")
    ("heights-from-dem-robust-threshold",
     po::value(&opt.heights_from_dem_robust_threshold)->default_value(0.5),
     "The robust threshold to use keep the triangulated points "
     "close to the DEM if specified via --heights-from-dem. This is applied after the "
     "point differences are multiplied by --heights-from-dem-weight. It should help with "
     "attenuating large height difference outliers. It is suggested to make this equal to "
     "--heights-from-dem-weight.")
    ("reference-dem",  po::value(&opt.ref_dem)->default_value(""),
     "If specified, intersect rays from matching pixels with this DEM, find the average, and constrain during optimization that rays keep on intersecting close to this point. This works even when the rays are almost parallel, but then consider using the option --forced-triangulation-distance. See also --reference-dem-weight and --reference-dem-robust-threshold.")
    ("reference-dem-weight", po::value(&opt.ref_dem_weight)->default_value(1.0),
     "Multiply the xyz differences for the --reference-dem option by this weight.")
    ("reference-dem-robust-threshold", po::value(&opt.ref_dem_robust_threshold)->default_value(0.5),
     "Use this robust threshold for the weighted xyz differences.")
    ("num-anchor-points", po::value(&opt.num_anchor_points)->default_value(0),
     "How many anchor points to create. They will be uniformly distributed "
     "across each input image. Only applies to linescan cameras.")
    ("anchor-weight", po::value(&opt.anchor_weight)->default_value(0.0),
     "How much weight to give to each anchor point. Anchor points are "
     "obtained by intersecting rays from initial cameras with the DEM given by "
     "--heights-from-dem. A larger weight will make it harder for "
     "the cameras to move, hence preventing unreasonable changes. "
     "Set also --anchor-weight and --anchor-dem.")
    ("anchor-dem",  po::value(&opt.anchor_dem)->default_value(""),
     "Use this DEM to create anchor points.")
    ("num-anchor-points-extra-lines",
     po::value(&opt.num_anchor_points_extra_lines)->default_value(0),
     "Start placing anchor points this many lines before first image line "
     "and after last image line.")
    ("rotation-weight", po::value(&opt.rotation_weight)->default_value(0.0),
     "A higher weight will penalize more deviations from the original camera orientations.")
    ("translation-weight", po::value(&opt.translation_weight)->default_value(0.0),
     "A higher weight will penalize more deviations from "
     "the original camera positions.")
    ("quat-norm-weight", po::value(&opt.quat_norm_weight)->default_value(1.0),
     "How much weight to give to the constraint that the norm of each quaternion must be 1.")
    ("roll-weight", po::value(&opt.roll_weight)->default_value(0.0),
     "A weight to penalize the deviation of camera roll orientation as measured from the "
     "along-track direction. Pass in a large value, such as 1e+5. This is best used only with "
     "linescan cameras created with sat_sim.")
    ("yaw-weight", po::value(&opt.yaw_weight)->default_value(0.0),
     "A weight to penalize the deviation of camera yaw orientation as measured from the "
     "along-track direction. Pass in a large value, such as 1e+5. This is best used only "
     "with linescan cameras created with sat_sim.")
    ("ip-side-filter-percent",  po::value(&opt.ip_edge_buffer_percent)->default_value(-1.0),
     "Remove matched IPs this percentage from the image left/right sides.")
    ("initial-camera-constraint", 
    po::bool_switch(&opt.initial_camera_constraint)->default_value(false),
     "When constraining roll and yaw, measure these not in the satellite along-track/ "
     "across-track/down coordinate system, but relative to the initial camera poses. This "
     "is experimental. Internally, the roll weight will then be applied to the camera pitch "
     "angle (rotation around the camera y axis), because the camera coordinate system is "
     "rotated by 90 degrees in the sensor plane relative to the satellite coordinate system. "
     "The goal is the same, to penalize deviations that are not aligned with satellite pitch.")
    ("forced-triangulation-distance",      po::value(&opt.forced_triangulation_distance)->default_value(-1),
     "When triangulation fails, for example, when input cameras are inaccurate, artificially create a triangulation point this far ahead of the camera, in units of meter.")     
    ;
  
    general_options.add(vw::GdalWriteOptionsDescription(opt));

  // TODO(oalexan1): This old option may need to be wiped given the newer
  // recent outlier filtering.
  asp::stereo_settings().ip_edge_buffer_percent = opt.ip_edge_buffer_percent;

  po::options_description positional("");
  positional.add_options()
    ("input-files", po::value(&opt.image_files));
  
  po::positional_options_description positional_desc;
  positional_desc.add("input-files", -1);

  std::string usage("<images> <cameras> -o <output prefix> [options]");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                            positional, positional_desc, usage,
                            allow_unregistered, unregistered);

  // Do this check first, as the output prefix is used below many times
  if (opt.out_prefix == "") 
    vw_throw(ArgumentErr() << "Must specify the output prefix.\n" << usage << "\n");

  // Create the output directory
  vw::create_out_dir(opt.out_prefix);

  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);

  // Set this before loading cameras, as jitter for DG can be modeled only with CSM
  // cameras.
  asp::stereo_settings().dg_use_csm = true;
  
  std::vector<std::string> inputs = opt.image_files;
  bool ensure_equal_sizes = true;
  asp::separate_images_from_cameras(inputs,
                                    opt.image_files, opt.camera_files, // outputs
                                    ensure_equal_sizes); 

  // This is needed when several images are acquired in quick succession
  // and we want to impose roll and yaw constraints given their orbital 
  // trajectory.
  asp::readGroupStructure(inputs, opt.orbital_groups);

  // Throw if there are duplicate camera file names.
  asp::check_for_duplicates(opt.image_files, opt.camera_files, opt.out_prefix);
  
  const int num_images = opt.image_files.size();
  
  // Sanity check
  if (opt.image_files.size() != opt.camera_files.size())
    vw_throw(ArgumentErr() << "Must have as many cameras as  have images.\n");
  
  if (opt.image_files.empty())
    vw_throw(ArgumentErr() << "Missing input image files.\n");
  
  if (opt.overlap_limit < 0)
    vw_throw(ArgumentErr() << "Must allow search for matches between "
             << "at least each image and its subsequent one.\n");
  
  // By default, try to match all of the images
  if (opt.overlap_limit == 0)
    opt.overlap_limit = opt.image_files.size();
  
  if (int(opt.match_files_prefix.empty()) + int(opt.clean_match_files_prefix.empty()) != 1) 
    vw_throw(ArgumentErr() << "Must specify precisely one of: --match-files-prefix, "
             << "--clean-match-files-prefix.\n");

  if (opt.max_init_reproj_error <= 0.0)
    vw_throw(ArgumentErr() << "Must have a positive --max-initial-reprojection-error.\n");

  if (!opt.heights_from_dem.empty() && !opt.ref_dem.empty()) 
    vw_throw(ArgumentErr() << "Cannot specify more than one of: --heights-from-dem "
             << "and --reference-dem.\n");

  if (opt.tri_weight < 0.0) 
    vw_throw(ArgumentErr() << "The value of --tri-weight must be non-negative.\n");

  if (opt.robust_threshold <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --robust-threshold must be positive.\n");

  if (opt.tri_robust_threshold <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --tri-robust-threshold must be positive.\n");
  
  if (opt.heights_from_dem_weight <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --heights-from-dem-weight must be positive.\n");
  
  if (opt.heights_from_dem_robust_threshold <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --heights-from-robust-threshold must be positive.\n");

  if (opt.ref_dem_weight <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --reference-dem-weight must be positive.\n");
  
  if (opt.ref_dem_robust_threshold <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --reference-dem-robust-threshold must be positive.\n");

  if (opt.rotation_weight < 0 || opt.translation_weight < 0)
    vw_throw(ArgumentErr() << "Rotation and translation weights must be non-negative.\n");
    
  if (opt.quat_norm_weight <= 0)
    vw_throw(ArgumentErr() << "Quaternion norm weight must be positive.\n");

  if (opt.roll_weight < 0.0)
    vw_throw(ArgumentErr() << "Roll weight must be non-negative.\n");

  if (opt.yaw_weight < 0.0)
    vw_throw(ArgumentErr() << "Yaw weight must be non-negative.\n");

  // Handle the roll/yaw constraint DEM
  if ((opt.roll_weight > 0 || opt.yaw_weight > 0) &&
     opt.heights_from_dem == "" && opt.ref_dem == "" && opt.anchor_dem == "")
      vw::vw_throw(ArgumentErr() << "Cannot use the roll/yaw constraint without a DEM. "
        << "Set either --heights-from-dem, --anchor-dem, or --reference-dem.\n");

  if (opt.num_anchor_points < 0)
    vw_throw(ArgumentErr() << "The number of anchor points must be non-negative.\n");

  if (opt.anchor_weight < 0)
    vw_throw(ArgumentErr() << "Anchor weight must be non-negative.\n");

  if (opt.anchor_weight > 0 && opt.anchor_dem.empty()) 
    vw::vw_throw(vw::ArgumentErr() << "If --anchor-weight is positive, set --anchor-dem.\n");
  
  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);
  
  return;
}

void compute_residuals(Options const& opt,
                       ceres::Problem & problem,
                       // Output
                       std::vector<double> & residuals) {

  double cost = 0.0;
  ceres::Problem::EvaluateOptions eval_options;
  eval_options.apply_loss_function = false;
  if (opt.single_threaded_cameras)
    eval_options.num_threads = 1; // ISIS must be single threaded!
  else
    eval_options.num_threads = opt.num_threads;
  
  problem.Evaluate(eval_options, &cost, &residuals, 0, 0);
}

void write_per_xyz_pixel_residuals(vw::ba::ControlNetwork const& cnet,
                                   std::string            const& residual_prefix,
                                   vw::cartography::Datum const& datum,
                                   std::set<int>          const& outliers,
                                   std::vector<double>    const& tri_points_vec,
                                   std::vector<double>    const& mean_pixel_residual_norm,
                                   std::vector<int>       const& pixel_residual_count) {
    
  std::string map_prefix = residual_prefix + "_pointmap";
  std::string output_path = map_prefix + ".csv";

  int num_tri_points = tri_points_vec.size() / NUM_XYZ_PARAMS;
  
  // Open the output file and write the header. TODO(oalexan1): See
  // if it is possible to integrate this with the analogous
  // bundle_adjust function.
  vw_out() << "Writing: " << output_path << std::endl;

  std::ofstream file;
  file.open(output_path.c_str());
  file.precision(17);
  file << "# lon, lat, height_above_datum, mean_residual, num_observations\n";
  file << "# " << datum << std::endl;

  // Write all the points to the file
  for (int ipt = 0; ipt < num_tri_points; ipt++) {

    if (outliers.find(ipt) != outliers.end() || pixel_residual_count[ipt] <= 0)
      continue; // Skip outliers
    
    // The final GCC coordinate of this point
    const double * tri_point = &tri_points_vec[0] + ipt * NUM_XYZ_PARAMS;
    Vector3 xyz(tri_point[0], tri_point[1], tri_point[2]);
    Vector3 llh = datum.cartesian_to_geodetic(xyz);
    
    std::string comment = "";
    if (cnet[ipt].type() == vw::ba::ControlPoint::GroundControlPoint)
      comment = " # GCP";
    file << llh[0] << ", " << llh[1] <<", " << llh[2] << ", "
         << mean_pixel_residual_norm[ipt] << ", "
         << pixel_residual_count[ipt] << comment << std::endl;
  }
  file.close();
}

void write_anchor_residuals(std::string            const& residual_prefix,
                            vw::cartography::Datum const& datum,
                            std::vector<Vector3>   const& anchor_xyz,
                            std::vector<double>    const& anchor_residual_norm) {
  
  std::string map_prefix = residual_prefix + "_anchor_points";
  std::string output_path = map_prefix + ".csv";
  vw_out() << "Writing: " << output_path << std::endl;
  std::ofstream file;
  file.open(output_path.c_str());
  file.precision(17);
  file << "# lon, lat, height_above_datum, anchor_residual_pixel_norm\n";
  file << "# " << datum << std::endl;

  for (size_t anchor_it = 0; anchor_it < anchor_xyz.size(); anchor_it++) {
    Vector3 llh = datum.cartesian_to_geodetic(anchor_xyz[anchor_it]);
    file << llh[0] <<", "<< llh[1] << ", " << llh[2] << ", "
         << anchor_residual_norm[anchor_it] << std::endl;
  }
  
  file.close();
}
                           
// TODO(oalexan1): Add here residuals for xyz discrepancy to DEM, if applicable
void save_residuals(std::string const& residual_prefix,
                    ceres::Problem & problem, Options const& opt,
                    vw::ba::ControlNetwork const& cnet,
                    vw::ba::CameraRelationNetwork<vw::ba::JFeature> const& crn,
                    bool have_dem, vw::cartography::Datum const& datum,
                    std::vector<double> const& tri_points_vec,
                    std::vector<Vector3> const& dem_xyz_vec,
                    std::set<int> const& outliers,
                    std::vector<double> const& weight_per_residual,
                    // These are needed for anchor points
                    std::vector<std::vector<Vector2>>                    const& pixel_vec,
                    std::vector<std::vector<boost::shared_ptr<Vector3>>> const& xyz_vec,
                    std::vector<std::vector<double*>>                    const& xyz_vec_ptr,
                    std::vector<std::vector<double>>                     const& weight_vec,
                    std::vector<std::vector<int>>                        const& isAnchor_vec) {
  
  // Compute the residuals before optimization
  std::vector<double> residuals;
  compute_residuals(opt, problem, residuals);
  if (residuals.size() != weight_per_residual.size()) 
    vw_throw(ArgumentErr() << "There must be as many residuals as weights for them.\n");

  //  Find the mean of all residuals corresponding to the same xyz point
  int num_tri_points = cnet.size();
  std::vector<double> mean_pixel_residual_norm(num_tri_points, 0.0);
  std::vector<int>    pixel_residual_count(num_tri_points, 0);
  std::vector<double> xyz_residual_norm; // This is unfinished logic
  if (have_dem)
    xyz_residual_norm.resize(num_tri_points, -1.0); // so we can ignore bad ones
  
  int ires = 0;
  for (int icam = 0; icam < (int)crn.size(); icam++) {
    for (auto fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++) {
      
      // The index of the 3D point
      int ipt = (**fiter).m_point_id;
      
      if (outliers.find(ipt) != outliers.end())
        continue; // Skip outliers

      // Norm of pixel residual
      double norm = norm_2(Vector2(residuals[ires + 0] / weight_per_residual[ires + 0],
                                   residuals[ires + 1] / weight_per_residual[ires + 1]));

      mean_pixel_residual_norm[ipt] += norm;
      pixel_residual_count[ipt]++;
      
      ires += PIXEL_SIZE; // Update for the next iteration
    }
  }

  // Average all pixel residuals for a given xyz
  for (int ipt = 0; ipt < num_tri_points; ipt++) {
    if (outliers.find(ipt) != outliers.end() || pixel_residual_count[ipt] <= 0)
      continue; // Skip outliers
    mean_pixel_residual_norm[ipt] /= pixel_residual_count[ipt];
  }

  // Save the residuals
  write_per_xyz_pixel_residuals(cnet, residual_prefix, datum, outliers,  
                                tri_points_vec, mean_pixel_residual_norm,  
                                pixel_residual_count);

  // Add residuals for anchor points. That is pass 1 from
  // addReprojectionErrors(). We imitate here the same logic for that
  // pass. We continue to increment the ires counter from above.
  std::vector<Vector3> anchor_xyz;
  std::vector<double> anchor_residual_norm;
  for (int pass = 1; pass < 2; pass++) {
    for (int icam = 0; icam < (int)crn.size(); icam++) {
      for (size_t ipix = 0; ipix < pixel_vec[icam].size(); ipix++) {

        Vector2 observation =  pixel_vec[icam][ipix];
        double * tri_point = xyz_vec_ptr[icam][ipix];
        double weight = weight_vec[icam][ipix];
        bool isAnchor = isAnchor_vec[icam][ipix];

        // Pass 0 is without anchor points, while pass 1 uses them.
        // Here we only do pass 1.
        if ((int)isAnchor != pass) 
          continue;

        if (weight != opt.anchor_weight)
          vw::vw_throw(vw::ArgumentErr() << "Expecting the weight to equal the anchor weight.\n");
        
        // Norm of pixel residual
        double norm = norm_2(Vector2(residuals[ires + 0] / weight_per_residual[ires + 0],
                                     residuals[ires + 1] / weight_per_residual[ires + 1]));
        norm /= weight; // Undo the weight, to recover the pixel norm
        
        ires += PIXEL_SIZE; // Update for the next iteration

        Vector3 xyz(tri_point[0], tri_point[1], tri_point[2]);
        anchor_xyz.push_back(xyz);
        anchor_residual_norm.push_back(norm);
      }
    }
  }
  write_anchor_residuals(residual_prefix, datum, anchor_xyz, anchor_residual_norm);
  
  // TODO(oalexan1): Add here per-camera median residuals.
  // TODO(oalexan1): Save the xyz residual norms as well.
  if (have_dem) {
    for (int ipt = 0; ipt < num_tri_points; ipt++) {

      Vector3 observation = dem_xyz_vec.at(ipt);
      if (outliers.find(ipt) != outliers.end() || observation == Vector3(0, 0, 0)) 
        continue; // outlier

      // This is a Vector3 residual 
      double norm = norm_2(Vector3(residuals[ires + 0] / weight_per_residual[ires + 0],
                                   residuals[ires + 1] / weight_per_residual[ires + 1],
                                   residuals[ires + 2] / weight_per_residual[ires + 2]));
      xyz_residual_norm[ipt] = norm;
      
      ires += NUM_XYZ_PARAMS; // Update for the next iteration
    }
  }

  // Ensure we did not process more residuals than what we have.
  // (Here we may not necessarily process all residuals.)
  if (ires > (int)residuals.size())
    vw_throw(ArgumentErr() << "More residuals found than expected.\n");

  return;
}


// Calc the time of first image line, last image line, elapsed time
// between these lines, and elapsed time per line.  This assumes a
// linear relationship between lines and time.
// TODO(oalexan1): This is fragile. Maybe it can be avoided.
void calcTimes(UsgsAstroLsSensorModel const* ls_model,
               double & earlier_line_time, double & later_line_time,
               double & elapsed_time, double & dt_per_line) {

  int numLines = ls_model->m_nLines;
  csm::ImageCoord imagePt;

  asp::toCsmPixel(vw::Vector2(0, 0), imagePt);
  earlier_line_time = ls_model->getImageTime(imagePt);

  asp::toCsmPixel(vw::Vector2(0, numLines - 1), imagePt);
  later_line_time = ls_model->getImageTime(imagePt);

  // See note in resampleModel().
  if (earlier_line_time > later_line_time)
    std::swap(earlier_line_time, later_line_time);
  
  elapsed_time = later_line_time - earlier_line_time;
  dt_per_line = elapsed_time / (numLines - 1.0);

  if (later_line_time <= earlier_line_time)
    vw::vw_throw(vw::ArgumentErr()
                 << "The time of the last line (in scanning order) must be larger than "
                 << "first line time.\n");
  
  return;
}

// Calculate the line index for first and last tabulated position.
// We always expect these to be less than first line index (0), and no less
// than last valid image line index (numLines - 1), respectively.
// TODO(oalexan1): This assumes a linear relationship between time and lines,
// which is fragile. At least need to check that this assumption is satisfied.
void calcFirstLastPositionLines(UsgsAstroLsSensorModel const* ls_model, 
                                double & beg_position_line, double & end_position_line) {

  double earlier_line_time = -1.0, later_line_time = -1.0, 
         elapsed_time = -1.0, dt_per_line = -1.0;
  calcTimes(ls_model, earlier_line_time, later_line_time, elapsed_time,  
               dt_per_line);
  
  // Find time of first and last tabulated position.
  double bt = ls_model->m_t0Ephem;
  double et = bt + (ls_model->m_positions.size()/NUM_XYZ_PARAMS - 1) * ls_model->m_dtEphem;

  // Use the equation: time = earlier_line_time + line * dt_per_line.
  // See note in resampleModel() about scan direction.
  beg_position_line = (bt - earlier_line_time) / dt_per_line;
  end_position_line = (et - earlier_line_time) / dt_per_line;

  // Sanity checks
  if (beg_position_line > 1e-3) // allow for rounding errors 
    vw::vw_throw(vw::ArgumentErr() << "Line of first tabulated position is "
                 << beg_position_line << ", which is after first image line, which is "
                 << 0 << ".\n");
  int numLines = ls_model->m_nLines;
  if (end_position_line < numLines - 1 - 1e-3)  // allow for rounding errors
    vw::vw_throw(vw::ArgumentErr() << "Line of last tabulated position is "
                 << end_position_line << ", which is before last image line, which is "
                 << numLines - 1 << ".\n");
}
  
// Calculate the line index for first and last tabulated orientation.
// We always expect these to be less than first line index (0), and no less
// than last valid image line index (numLines - 1), respectively.
void calcFirstLastOrientationLines(UsgsAstroLsSensorModel const* ls_model, 
                                   double & beg_orientation_line, double & end_orientation_line) {

  double earlier_line_time = -1.0, later_line_time = -1.0, 
         elapsed_time = -1.0, dt_per_line = -1.0;
  calcTimes(ls_model, earlier_line_time, later_line_time, elapsed_time,  
               dt_per_line);
  
  // Find time of first and last tabulated orientation.
  double bt = ls_model->m_t0Quat;
  double et = bt + (ls_model->m_quaternions.size()/NUM_QUAT_PARAMS - 1) * ls_model->m_dtQuat;
  
  // Use the equation: time = earlier_line_time + line * dt_per_line.
  beg_orientation_line = (bt - earlier_line_time) / dt_per_line;
  end_orientation_line = (et - earlier_line_time) / dt_per_line;

  // Sanity checks
  if (beg_orientation_line > 1e-3) // allow for rounding errors 
    vw::vw_throw(vw::ArgumentErr() << "Line of first tabulated orientation is "
                 << beg_orientation_line << ", which is after first image line, which is "
                   << 0 << ".\n");
  int numLines = ls_model->m_nLines;
  if (end_orientation_line < numLines - 1 - 1e-3)  // allow for rounding errors
    vw::vw_throw(vw::ArgumentErr() << "Line of last tabulated orientation is "
                 << end_orientation_line << ", which is before last image line, which is "
                   << numLines - 1 << ".\n");
}

// TODO(oalexan1): Move the function below out of here, to CsmUtils.cc.
// The provided tabulated positions, velocities and quaternions may be too few,
// so resample them with --num-lines-per-position and --num-lines-per-orientation,
// if those are set. Throughout this function the lines are indexed in the order
// they are acquired, which can be the reverse of the order they are eventually
// stored in the file if the scan direction is reverse.
void resampleModel(Options const& opt, UsgsAstroLsSensorModel * ls_model) {
  
  // The positions and quaternions can go way beyond the valid range of image lines,
  // so need to estimate how many of them are within the range.
  
  int numLines = ls_model->m_nLines;
  vw_out() << "Number of lines: " << numLines << ".\n";

  double earlier_line_time = -1.0, later_line_time = -1.0, elapsed_time = -1.0, dt_per_line = -1.0;
  calcTimes(ls_model, earlier_line_time, later_line_time, elapsed_time,  
            dt_per_line);

  // Line index of first and last tabulated position
  double beg_position_line = -1.0, end_position_line = -1.0;
  calcFirstLastPositionLines(ls_model, beg_position_line, end_position_line);
  vw_out() << std::setprecision (17) << "Line of first and last tabulated position: "
           << beg_position_line << ' ' << end_position_line << "\n";

  // Line index of first and last tabulated orientation
  double beg_orientation_line = -1.0, end_orientation_line = -1.0;
  calcFirstLastOrientationLines(ls_model, beg_orientation_line, end_orientation_line);
  vw_out() << std::setprecision (17) << "Line of first and last tabulated orientation: "
           << beg_orientation_line << ' ' << end_orientation_line << "\n";

  double numInputLinesPerPosition = (numLines - 1) * ls_model->m_dtEphem / elapsed_time;
  double numInputLinesPerOrientation = (numLines - 1) * ls_model->m_dtQuat / elapsed_time;
  vw_out() << "Number of image lines per input position: "
           << round(numInputLinesPerPosition) << "\n";
  vw_out() << "Number of image lines per input orientation: "
           << round(numInputLinesPerOrientation) << "\n";

  if (opt.num_lines_per_position > 0) {
    // Resample in such a way that first and last samples are preserved. This is tricky.
    double posFactor = double(numInputLinesPerPosition) / double(opt.num_lines_per_position);
    if (posFactor <= 0.0)
      vw::vw_throw(vw::ArgumentErr() << "Invalid image.\n");

    std::cout << "pos factor " << posFactor << std::endl;
    int numOldMeas = ls_model->m_numPositions / NUM_XYZ_PARAMS;
    int numNewMeas = round(posFactor * (numOldMeas - 1.0)) + 1; // careful here
    numNewMeas = std::max(numNewMeas, 2);
    std::cout << "num old pos meas " << numOldMeas << std::endl;
    std::cout << "num new pos meas " << numNewMeas << std::endl;

    posFactor = double(numNewMeas - 1.0) / double(numOldMeas - 1.0);
    double currDtEphem = ls_model->m_dtEphem / posFactor;
    double numLinesPerPosition = (numLines - 1.0) * currDtEphem / elapsed_time;
    vw_out() << "Resampled number of lines per position: "
             << numLinesPerPosition << "\n";
    std::vector<double> positions(NUM_XYZ_PARAMS * numNewMeas, 0);
    std::vector<double> velocities(NUM_XYZ_PARAMS * numNewMeas, 0);
    for (int ipos = 0; ipos < numNewMeas; ipos++) {
      double time = ls_model->m_t0Ephem + ipos * currDtEphem;
      asp::interpPositions(ls_model, time, &positions[NUM_XYZ_PARAMS * ipos]);
      asp::interpVelocities(ls_model, time, &velocities[NUM_XYZ_PARAMS * ipos]);
    }
    
    // Overwrite in the model. Time of first tabulated position does not change.
    ls_model->m_dtEphem = currDtEphem;
    ls_model->m_numPositions = positions.size();
    ls_model->m_positions = positions;
    ls_model->m_velocities = velocities;

    // Sanity check
    double new_beg_position_line = -1.0, new_end_position_line = -1.0;
    calcFirstLastPositionLines(ls_model, new_beg_position_line, new_end_position_line);
    if (std::abs(beg_position_line - new_beg_position_line) > 1.0e-3 ||
        std::abs(end_position_line - new_end_position_line) > 1.0e-3)
      vw::vw_throw(vw::ArgumentErr() << "Bookkeeping failure. Resampling was done "
                   << "without preserving first and last tabulated position time.\n");
  }

  if (opt.num_lines_per_orientation > 0) {
    // Resample in such a way that first and last samples are preserved. This is tricky.
    double posFactor = double(numInputLinesPerOrientation) / double(opt.num_lines_per_orientation);
    if (posFactor <= 0.0)
      vw::vw_throw(vw::ArgumentErr() << "Invalid image.\n");

    int numOldMeas = ls_model->m_numQuaternions / NUM_QUAT_PARAMS;
    int numNewMeas = round(posFactor * (numOldMeas - 1.0)) + 1; // careful here
    numNewMeas = std::max(numNewMeas, 2);

    std::cout << "quat factor " << posFactor << std::endl;
    std::cout << "num old quat meas " << numOldMeas << std::endl;
    std::cout << "num new quat meas " << numNewMeas << std::endl;
    
    posFactor = double(numNewMeas - 1.0) / double(numOldMeas - 1.0);
    double currDtQuat = ls_model->m_dtQuat / posFactor;
    double numLinesPerOrientation = (numLines - 1.0) * currDtQuat / elapsed_time;
    vw_out() << "Resampled number of lines per orientation: "
             << numLinesPerOrientation << "\n";
    std::vector<double> quaternions(NUM_QUAT_PARAMS * numNewMeas, 0);
    for (int ipos = 0; ipos < numNewMeas; ipos++) {
      double time = ls_model->m_t0Quat + ipos * currDtQuat;
      asp::interpQuaternions(ls_model, time, &quaternions[NUM_QUAT_PARAMS * ipos]);
    }
    
    // Overwrite in the model. Time of first tabulated orientation does not change.
    ls_model->m_dtQuat = currDtQuat;
    ls_model->m_numQuaternions = quaternions.size();
    ls_model->m_quaternions = quaternions;

    // Sanity check
    double new_beg_orientation_line = -1.0, new_end_orientation_line = -1.0;
    calcFirstLastOrientationLines(ls_model, new_beg_orientation_line, new_end_orientation_line);
    if (std::abs(beg_orientation_line - new_beg_orientation_line) > 1.0e-3 ||
        std::abs(end_orientation_line - new_end_orientation_line) > 1.0e-3)
      vw::vw_throw(vw::ArgumentErr() << "Bookkeeping failure. Resampling was done "
                   << "without preserving first and last tabulated orientation time.\n");
  }

  return;
}

// Calculate a set of anchor points uniformly distributed over the image
// Will use opt.num_anchor_points_extra_lines.
void calcAnchorPoints(Options                              const  & opt,
                      ImageViewRef<PixelMask<double>>               interp_anchor_dem,
                      vw::cartography::GeoReference         const & anchor_georef,
                      std::vector<asp::CsmModel*>           const & csm_models,
                      // Append to these, they already have entries
                      std::vector<std::vector<Vector2>>                    & pixel_vec,
                      std::vector<std::vector<boost::shared_ptr<Vector3>>> & xyz_vec,
                      std::vector<std::vector<double*>>                    & xyz_vec_ptr,
                      std::vector<std::vector<double>>                     & weight_vec,
                      std::vector<std::vector<int>>                        & isAnchor_vec) {

  if (opt.num_anchor_points <= 0)
    vw::vw_throw(vw::ArgumentErr() << "Expecting a positive number of anchor points.\n");

  int extra = opt.num_anchor_points_extra_lines;
    
  int num_cams = csm_models.size();
  for (int icam = 0; icam < num_cams; icam++) {

    UsgsAstroLsSensorModel * ls_model
      = dynamic_cast<UsgsAstroLsSensorModel*>((csm_models[icam]->m_gm_model).get());
    if (ls_model == NULL)
      continue; // anchor points not implemented for Frame cameras

    // Use int64 and double to avoid int32 overflow
    std::int64_t numLines   = ls_model->m_nLines;
    std::int64_t numSamples = ls_model->m_nSamples;
    double area = double(numSamples) * double(numLines + 2 * extra);
    double bin_len = sqrt(area/double(opt.num_anchor_points));
    bin_len = std::max(bin_len, 1.0);
    int lenx = ceil(double(numSamples) / bin_len); lenx = std::max(1, lenx);
    int leny = ceil(double(numLines + 2 * extra) / bin_len); leny = std::max(1, leny);

    std::int64_t numAnchorPoints = 0;
    for (int binx = 0; binx <= lenx; binx++) {
      double posx = binx * bin_len;
      for (int biny = 0; biny <= leny; biny++) {
        double posy = biny * bin_len - extra;
        
        if (posx > numSamples - 1 || posy < -extra || posy > numLines - 1 + extra) 
          continue;
        
        Vector2 pix(posx, posy);
        Vector3 xyz_guess(0, 0, 0);
        
        bool treat_nodata_as_zero = false;
        bool has_intersection = false;
        double height_error_tol = 0.001; // 1 mm should be enough
        double max_abs_tol      = 1e-14; // abs cost fun change b/w iterations
        double max_rel_tol      = 1e-14;
        int num_max_iter        = 50;   // Using many iterations can be very slow
          
        Vector3 dem_xyz = vw::cartography::camera_pixel_to_dem_xyz
          (opt.camera_models[icam]->camera_center(pix),
           opt.camera_models[icam]->pixel_to_vector(pix),
           interp_anchor_dem, anchor_georef, treat_nodata_as_zero, has_intersection,
           height_error_tol, max_abs_tol, max_rel_tol, num_max_iter, xyz_guess);

        if (!has_intersection) 
          continue;

        Vector2 pix_out;
        try {
          pix_out = opt.camera_models[icam]->point_to_pixel(dem_xyz);
        } catch (...) {
          continue;
        }
        
        if (norm_2(pix - pix_out) > 10 * height_error_tol)
          continue; // this is likely a bad point

        pixel_vec[icam].push_back(pix);
        weight_vec[icam].push_back(opt.anchor_weight);
        isAnchor_vec[icam].push_back(1);

        // Create a shared_ptr as we need a pointer per the api to use later
        xyz_vec[icam].push_back(boost::shared_ptr<Vector3>(new Vector3()));
        Vector3 & xyz = *xyz_vec[icam].back().get(); // alias to the element we just made
        xyz = dem_xyz; // copy the value, but the pointer does not change
        xyz_vec_ptr[icam].push_back(&xyz[0]); // keep the pointer to the first element
        numAnchorPoints++;
      }   
    }

    vw_out() << std::endl;
    vw_out() << "Image file: " << opt.image_files[icam] << std::endl;
    vw_out() << "Lines and samples: " << numLines << ' ' << numSamples << std::endl;
    vw_out() << "Num anchor points per image: " << numAnchorPoints     << std::endl;
  }   
}

// Add the linescan model reprojection error to the cost function
void addLsReprojectionErr(Options          const & opt,
                          UsgsAstroLsSensorModel * ls_model,
                          vw::Vector2      const & observation,
                          double                 * tri_point,
                          double                   weight,
                          ceres::Problem         & problem) {

  // Must grow the number of quaternions and positions a bit
  // because during optimization the 3D point and corresponding
  // pixel may move somewhat.
  double line_extra = opt.max_init_reproj_error + 5.0; // add some more just in case
  csm::ImageCoord imagePt1, imagePt2;
  asp::toCsmPixel(observation - Vector2(0.0, line_extra), imagePt1);
  asp::toCsmPixel(observation + Vector2(0.0, line_extra), imagePt2);
  double time1 = ls_model->getImageTime(imagePt1);
  double time2 = ls_model->getImageTime(imagePt2);

  // Handle quaternions. We follow closely the conventions for UsgsAstroLsSensorModel.
  int numQuatPerObs = 8; // Max num of quaternions used in pose interpolation 
  int numQuat       = ls_model->m_quaternions.size() / NUM_QUAT_PARAMS;
  double quatT0     = ls_model->m_t0Quat;
  double quatDt     = ls_model->m_dtQuat;

  // Starting and ending quat index (ending is exclusive). Based on lagrangeInterp().
  int qindex1      = static_cast<int>((time1 - quatT0) / quatDt);
  int qindex2      = static_cast<int>((time2 - quatT0) / quatDt);
  int begQuatIndex = std::min(qindex1, qindex2) - numQuatPerObs / 2 + 1;
  int endQuatIndex = std::max(qindex1, qindex2) + numQuatPerObs / 2 + 1;

  // Keep in bounds
  begQuatIndex = std::max(0, begQuatIndex);
  endQuatIndex = std::min(endQuatIndex, numQuat);
  if (begQuatIndex >= endQuatIndex) {
    // Must not happen 
    vw::vw_throw(vw::ArgumentErr() << "Book-keeping error for quaternions for pixel: " 
      << observation << ". Check your image dimensions and compare "
      << "with the camera file.\n"); 
  }

  // Same for positions
  int numPosPerObs = 8;
  int numPos       = ls_model->m_positions.size() / NUM_XYZ_PARAMS;
  double posT0     = ls_model->m_t0Ephem;
  double posDt     = ls_model->m_dtEphem;

  // Starting and ending pos index (ending is exclusive). Based on lagrangeInterp().
  int pindex1 = static_cast<int>((time1 - posT0) / posDt);
  int pindex2 = static_cast<int>((time2 - posT0) / posDt);
  int begPosIndex = std::min(pindex1, pindex2) - numPosPerObs / 2 + 1;
  int endPosIndex = std::max(pindex1, pindex2) + numPosPerObs / 2 + 1;

  // Keep in bounds
  begPosIndex = std::max(0, begPosIndex);
  endPosIndex = std::min(endPosIndex, numPos);
  if (begPosIndex >= endPosIndex) // Must not happen 
    vw_throw(ArgumentErr() << "Book-keeping error for positions for pixel: " 
      << observation << ". Check your image dimensions and compare "
      << "with the camera file.\n");

  ceres::CostFunction* pixel_cost_function =
    LsPixelReprojErr::Create(observation, weight, ls_model,
                              begQuatIndex, endQuatIndex,
                              begPosIndex, endPosIndex);
  ceres::LossFunction* pixel_loss_function = new ceres::CauchyLoss(opt.robust_threshold);

  // The variable of optimization are camera quaternions and positions stored in the
  // camera models, and the triangulated point.
  std::vector<double*> vars;
  for (int it = begQuatIndex; it < endQuatIndex; it++)
    vars.push_back(&ls_model->m_quaternions[it * NUM_QUAT_PARAMS]);
  for (int it = begPosIndex; it < endPosIndex; it++)
    vars.push_back(&ls_model->m_positions[it * NUM_XYZ_PARAMS]);
  vars.push_back(tri_point);
  problem.AddResidualBlock(pixel_cost_function, pixel_loss_function, vars);

  return;   
}

// Add the frame camera model reprojection error to the cost function
void addFrameReprojectionErr(Options             const & opt,
                             UsgsAstroFrameSensorModel * frame_model,
                             vw::Vector2         const & observation,
                             double                    * frame_params,
                             double                    * tri_point,
                             double                      weight,
                             ceres::Problem            & problem) {

  ceres::CostFunction* pixel_cost_function =
    FramePixelReprojErr::Create(observation, weight, frame_model);
  ceres::LossFunction* pixel_loss_function = new ceres::CauchyLoss(opt.robust_threshold);

  // The variable of optimization are camera positions and quaternion stored 
  // in frame_cam_params, in this order, and the triangulated point.
  // This is different from the linescan model, where we can directly access
  // these quantities inside the model, so they need not be stored separately.
  std::vector<double*> vars;
  vars.push_back(&frame_params[0]);              // positions start here
  vars.push_back(&frame_params[NUM_XYZ_PARAMS]); // quaternions start here
  vars.push_back(tri_point);
  problem.AddResidualBlock(pixel_cost_function, pixel_loss_function, vars);

  return;   
}

void addReprojectionErrors
(Options                                              const & opt,
 vw::ba::CameraRelationNetwork<vw::ba::JFeature>      const & crn,
 std::vector<std::vector<Vector2>>                    const & pixel_vec,
 std::vector<std::vector<boost::shared_ptr<Vector3>>> const & xyz_vec,
 std::vector<std::vector<double*>>                    const & xyz_vec_ptr,
 std::vector<std::vector<double>>                     const & weight_vec,
 std::vector<std::vector<int>>                        const & isAnchor_vec,
 std::vector<asp::CsmModel*>                          const & csm_models,
 // Outputs
 std::vector<double>                                        & frame_params,
 std::vector<double>                                        & weight_per_residual, // append
 ceres::Problem                                             & problem) {

  // Do here two passes, first for non-anchor points and then for anchor ones.
  // This way it is easier to do the bookkeeping when saving the residuals.
  // Note: The same motions as here are repeated in save_residuals().
  for (int pass = 0; pass < 2; pass++) {
    for (int icam = 0; icam < (int)crn.size(); icam++) {

      for (size_t ipix = 0; ipix < pixel_vec[icam].size(); ipix++) {

        Vector2 observation = pixel_vec[icam][ipix];
        double * tri_point  = xyz_vec_ptr[icam][ipix];
        double weight       = weight_vec[icam][ipix];
        bool isAnchor       = isAnchor_vec[icam][ipix];

        // Pass 0 is without anchor points, while pass 1 uses them
        if ((int)isAnchor != pass) 
          continue;

        // We can have linescan or frame cameras 
        UsgsAstroLsSensorModel * ls_model
          = dynamic_cast<UsgsAstroLsSensorModel*>((csm_models[icam]->m_gm_model).get());
        UsgsAstroFrameSensorModel * frame_model
          = dynamic_cast<UsgsAstroFrameSensorModel*>((csm_models[icam]->m_gm_model).get());
  
        // Note how for the frame model we pass the frame_params for the current camera.
        if (ls_model != NULL)
          addLsReprojectionErr(opt, ls_model, observation, tri_point, weight, problem);
        else if (frame_model != NULL)
          addFrameReprojectionErr(opt, frame_model, observation, 
              &frame_params[icam * (NUM_XYZ_PARAMS + NUM_QUAT_PARAMS)],
              tri_point, weight, problem);                   
        else
          vw::vw_throw(vw::ArgumentErr() << "Unknown camera model.\n");

        // Two residuals were added. Save the corresponding weights.
        for (int c = 0; c < PIXEL_SIZE; c++)
          weight_per_residual.push_back(weight);

        // Anchor points are fixed by definition. They try to prevent
        // the cameras from moving too much from original poses.
        if (isAnchor) 
          problem.SetParameterBlockConstant(tri_point);
      }
    }
  }
}

// Add the constraint based on DEM
void addDemConstraint
(Options                                              const& opt,
 std::vector<std::vector<boost::shared_ptr<Vector3>>> const& xyz_vec,
 std::vector<std::vector<double*>>                    const& xyz_vec_ptr,
 std::vector<vw::Vector3>                             const& dem_xyz_vec,
 std::set<int>                                        const& outliers,
 vw::ba::ControlNetwork                               const& cnet,
 // Outputs
 std::vector<double>                                       & tri_points_vec,
 std::vector<double>                                       & weight_per_residual, // append
 ceres::Problem                                            & problem) {
  
  double xyz_weight = -1.0, xyz_threshold = -1.0;
    
  if (!opt.heights_from_dem.empty()) {
    xyz_weight = opt.heights_from_dem_weight;
    xyz_threshold = opt.heights_from_dem_robust_threshold;
  } else if (!opt.ref_dem.empty()) {
    xyz_weight = opt.ref_dem_weight;
    xyz_threshold = opt.ref_dem_robust_threshold;
  } else {
    vw::vw_throw(vw::ArgumentErr() << "No input DEM was provided.\n");
  }
  
  if (dem_xyz_vec.size() != cnet.size()) 
    vw_throw(ArgumentErr() << "Must have as many xyz computed from DEM as xyz "
             << "triangulated from match files.\n");
  if (xyz_weight <= 0 || xyz_threshold <= 0)
    vw_throw(ArgumentErr() << "Detected invalid robust threshold or weights.\n");

  int num_tri_points = cnet.size();
  for (int ipt = 0; ipt < num_tri_points; ipt++) {
      
    if (cnet[ipt].type() == vw::ba::ControlPoint::GroundControlPoint)
      vw_throw(ArgumentErr() << "Found GCP where not expecting any.\n");

    // Note that we get tri points from dem_xyz_vec, based on the input DEM
    Vector3 observation = dem_xyz_vec.at(ipt);
    if (outliers.find(ipt) != outliers.end() || observation == Vector3(0, 0, 0)) 
      continue; // outlier
      
    ceres::CostFunction* xyz_cost_function = weightedXyzError::Create(observation, xyz_weight);
    ceres::LossFunction* xyz_loss_function = new ceres::CauchyLoss(xyz_threshold);
    double * tri_point = &tri_points_vec[0] + ipt * NUM_XYZ_PARAMS;

    // Add cost function
    problem.AddResidualBlock(xyz_cost_function, xyz_loss_function, tri_point);

    for (int c = 0; c < NUM_XYZ_PARAMS; c++)
      weight_per_residual.push_back(xyz_weight);
  }
}

// Add the constraint to keep triangulated points close to initial values
// This does not need a DEM or alignment
void addTriConstraint
(Options                                              const& opt,
 std::set<int>                                        const& outliers,
 vw::ba::ControlNetwork                               const& cnet,
 // Outputs
 std::vector<double>                                       & tri_points_vec,
 std::vector<double>                                       & weight_per_residual, // append
 ceres::Problem                                            & problem) {

  int num_tri_points = cnet.size();
  for (int ipt = 0; ipt < num_tri_points; ipt++) {
    if (cnet[ipt].type() == vw::ba::ControlPoint::GroundControlPoint ||
        cnet[ipt].type() == vw::ba::ControlPoint::PointFromDem)
      continue; // Skip GCPs and height-from-dem points which have their own constraint

    if (outliers.find(ipt) != outliers.end()) 
      continue; // skip outliers
      
    double * tri_point = &tri_points_vec[0] + ipt * NUM_XYZ_PARAMS;
      
    // Use as constraint the initially triangulated point
    vw::Vector3 observation(tri_point[0], tri_point[1], tri_point[2]);

    ceres::CostFunction* cost_function = weightedXyzError::Create(observation, opt.tri_weight);
    ceres::LossFunction* loss_function = new ceres::CauchyLoss(opt.tri_robust_threshold);
    problem.AddResidualBlock(cost_function, loss_function, tri_point);
    
    for (int c = 0; c < NUM_XYZ_PARAMS; c++)
      weight_per_residual.push_back(opt.tri_weight);
      
  } // End loop through xyz
}

void addQuatNormRotationTranslationConstraints(
    Options                                         const & opt,
    std::set<int>                                   const & outliers,
    vw::ba::CameraRelationNetwork<vw::ba::JFeature> const & crn,
    std::vector<asp::CsmModel*>                     const & csm_models,
    // Outputs
    std::vector<double>                                   & frame_params,
    std::vector<double>                                   & tri_points_vec,
    std::vector<double>                                   & weight_per_residual, // append
    ceres::Problem                                        & problem) {
  
  // Constrain the rotations
  // TODO(oalexan1): Make this a standalone function
  if (opt.rotation_weight > 0.0) {
    for (int icam = 0; icam < (int)crn.size(); icam++) {

      UsgsAstroLsSensorModel * ls_model
        = dynamic_cast<UsgsAstroLsSensorModel*>((csm_models[icam]->m_gm_model).get());
      UsgsAstroFrameSensorModel * frame_model
        = dynamic_cast<UsgsAstroFrameSensorModel*>((csm_models[icam]->m_gm_model).get());

      if (ls_model != NULL) {
        // There are multiple quaternion parameters per camera
        int numQuat = ls_model->m_quaternions.size() / NUM_QUAT_PARAMS;
        for (int iq = 0; iq < numQuat; iq++) {
          ceres::CostFunction* rotation_cost_function
            = weightedRotationError::Create(&ls_model->m_quaternions[iq * NUM_QUAT_PARAMS],
                                            opt.rotation_weight);
          // We use no loss function, as the quaternions have no outliers
          ceres::LossFunction* rotation_loss_function = NULL;
          problem.AddResidualBlock(rotation_cost_function, rotation_loss_function,
                                  &ls_model->m_quaternions[iq * NUM_QUAT_PARAMS]);
          
          for (int c = 0; c < NUM_QUAT_PARAMS; c++)
            weight_per_residual.push_back(opt.rotation_weight);
        }

      } else if (frame_model != NULL) {
        // There is one quaternion per camera, stored after the translation
        double * curr_params = &frame_params[icam * (NUM_XYZ_PARAMS + NUM_QUAT_PARAMS)];
          
        // Copy from curr_params the initial quaternion
        ceres::CostFunction* rotation_cost_function
          = weightedRotationError::Create(&curr_params[NUM_XYZ_PARAMS], // quat starts here
                                          opt.rotation_weight);
        // Pass the quaternion to optimize to the problem                                  
        // We use no loss function, as the quaternions have no outliers
        ceres::LossFunction* rotation_loss_function = NULL;
        problem.AddResidualBlock(rotation_cost_function, rotation_loss_function,
                                &curr_params[NUM_XYZ_PARAMS]); // quat starts here
        
        for (int c = 0; c < NUM_QUAT_PARAMS; c++)
          weight_per_residual.push_back(opt.rotation_weight);
      } else {
         vw::vw_throw(vw::ArgumentErr() << "Unknown camera model.\n");
      }

    } // end loop through cameras
  }

  // Constrain the translations
  // TODO(oalexan1): Make this a standalone function
  if (opt.translation_weight > 0.0) {
    for (int icam = 0; icam < (int)crn.size(); icam++) {

      UsgsAstroLsSensorModel * ls_model
        = dynamic_cast<UsgsAstroLsSensorModel*>((csm_models[icam]->m_gm_model).get());
      UsgsAstroFrameSensorModel * frame_model
        = dynamic_cast<UsgsAstroFrameSensorModel*>((csm_models[icam]->m_gm_model).get());

      if (ls_model != NULL) {

        int numPos = ls_model->m_positions.size() / NUM_XYZ_PARAMS;
        for (int ip = 0; ip < numPos; ip++) {
          ceres::CostFunction* translation_cost_function
            = weightedTranslationError::Create(&ls_model->m_positions[ip * NUM_XYZ_PARAMS],
                                            opt.translation_weight);
          // We use no loss function, as the positions have no outliers
          ceres::LossFunction* translation_loss_function = NULL;
          problem.AddResidualBlock(translation_cost_function, translation_loss_function,
                                  &ls_model->m_positions[ip * NUM_XYZ_PARAMS]);
          
          for (int c = 0; c < NUM_XYZ_PARAMS; c++)
            weight_per_residual.push_back(opt.translation_weight);
        }

      } else if (frame_model != NULL) {

        // There is only one position per camera
        double * curr_params = &frame_params[icam * (NUM_XYZ_PARAMS + NUM_QUAT_PARAMS)];
        // we will copy from curr_params the initial position
        ceres::CostFunction* translation_cost_function
          = weightedTranslationError::Create(&curr_params[0], // translation starts here
                                             opt.translation_weight);

        // We use no loss function, as the positions have no outliers
        ceres::LossFunction* translation_loss_function = NULL;
        problem.AddResidualBlock(translation_cost_function, translation_loss_function,
                                &curr_params[0]); // translation starts here
        
        for (int c = 0; c < NUM_XYZ_PARAMS; c++)
          weight_per_residual.push_back(opt.translation_weight);

      } else {
         vw::vw_throw(vw::ArgumentErr() << "Unknown camera model.\n");
      }

    }
  }

  // Try to make the norm of quaternions be close to 1
  // TODO(oalexan1): Make this a standalone function
  if (opt.quat_norm_weight > 0.0) {
    for (int icam = 0; icam < (int)crn.size(); icam++) {

      UsgsAstroLsSensorModel * ls_model
        = dynamic_cast<UsgsAstroLsSensorModel*>((csm_models[icam]->m_gm_model).get());
      UsgsAstroFrameSensorModel * frame_model
        = dynamic_cast<UsgsAstroFrameSensorModel*>((csm_models[icam]->m_gm_model).get());

      if (ls_model != NULL) {

        int numQuat = ls_model->m_quaternions.size() / NUM_QUAT_PARAMS;
        for (int iq = 0; iq < numQuat; iq++) {
          ceres::CostFunction* quat_norm_cost_function
            = weightedQuatNormError::Create(opt.quat_norm_weight);
          // We use no loss function, as the quaternions have no outliers
          ceres::LossFunction* quat_norm_loss_function = NULL;
          problem.AddResidualBlock(quat_norm_cost_function, quat_norm_loss_function,
                                  &ls_model->m_quaternions[iq * NUM_QUAT_PARAMS]);
          
          weight_per_residual.push_back(opt.quat_norm_weight); // 1 single residual
        }

      } else if (frame_model != NULL) {

        // There is one quaternion per camera, stored after the translation
        double * curr_params = &frame_params[icam * (NUM_XYZ_PARAMS + NUM_QUAT_PARAMS)];

        ceres::CostFunction* quat_norm_cost_function
          = weightedQuatNormError::Create(opt.quat_norm_weight);
        // We use no loss function, as the quaternions have no outliers
        ceres::LossFunction* quat_norm_loss_function = NULL;
        problem.AddResidualBlock(quat_norm_cost_function, quat_norm_loss_function,
                                &curr_params[NUM_XYZ_PARAMS]); // quat starts here
        
        weight_per_residual.push_back(opt.quat_norm_weight); // 1 single residual

      } else {
         vw::vw_throw(vw::ArgumentErr() << "Unknown camera model.\n");
      }
    }
  }
}

// Add roll / yaw constraints. For linescan, use the whole set of samples for given
// camera model. For frame cameras, use the trajectory of all cameras in the same orbital
// group as the current camera.
void addRollYawConstraint
   (Options                                         const& opt,
    vw::ba::CameraRelationNetwork<vw::ba::JFeature> const& crn,
    std::vector<asp::CsmModel*>                     const& csm_models,
    vw::cartography::GeoReference                   const& georef,
    // Outputs (append to residual)
    std::vector<double>                                  & frame_params,
    std::vector<double>                                  & weight_per_residual,
    ceres::Problem                                       & problem) {
  
  if (opt.roll_weight <= 0.0 && opt.yaw_weight <= 0.0)
     vw::vw_throw(vw::ArgumentErr() 
         << "addRollYawConstraint: The roll or yaw weight must be positive.\n");

  int num_cams = crn.size();

  // Frame cameras can be grouped by orbital portion. Ensure that all cameras
  // belong to a group.
  if (num_cams != int(opt.orbital_groups.size()))
    vw::vw_throw(vw::ArgumentErr() 
         << "addRollYawConstraint: Failed to add each input camera to an orbital group.\n");

  // Create the orbital trajectory for each group of frame cameras
  std::map<int, std::vector<double>> orbital_group_positions;
  std::map<int, std::vector<double>> orbital_group_quaternions;
  formPositionQuatVecPerGroup(opt.orbital_groups, csm_models, 
    orbital_group_positions, orbital_group_quaternions); // outputs

  for (int icam = 0; icam < num_cams; icam++) {

    UsgsAstroLsSensorModel * ls_model
      = dynamic_cast<UsgsAstroLsSensorModel*>((csm_models[icam]->m_gm_model).get());
    UsgsAstroFrameSensorModel * frame_model
      = dynamic_cast<UsgsAstroFrameSensorModel*>((csm_models[icam]->m_gm_model).get());

    if (ls_model != NULL) {
      // Linescan cameras. Use the full sequence of cameras in the model
      // to enforce the roll/yaw constraint for each camera in the sequence.
      int numQuat = ls_model->m_quaternions.size() / NUM_QUAT_PARAMS;

      // Make positions one-to-one with quaternions
      std::vector<double> interp_positions;
      asp::orbitInterpExtrap(ls_model, georef, interp_positions);
      
      for (int iq = 0; iq < numQuat; iq++) {
        ceres::CostFunction* roll_yaw_cost_function
          = weightedRollYawError::Create(interp_positions,
                                         ls_model->m_quaternions,
                                         georef, iq,
                                         opt.roll_weight, opt.yaw_weight, 
                                         opt.initial_camera_constraint);

        // We use no loss function, as the quaternions have no outliers
        ceres::LossFunction* roll_yaw_loss_function = NULL;
        problem.AddResidualBlock(roll_yaw_cost_function, roll_yaw_loss_function,
                                &ls_model->m_quaternions[iq * NUM_QUAT_PARAMS]);
        // The recorded weight should not be 0 as we will divide by it
        weight_per_residual.push_back(opt.roll_weight || 1.0);
        weight_per_residual.push_back(opt.yaw_weight  || 1.0);
      } // end loop through quaternions for given camera
    
    } else if (frame_model != NULL) {
      // Frame cameras. Use the positions and quaternions of the cameras
      // in the same orbital group to enforce the roll/yaw constraint for
      // each camera in the group.
      auto it = opt.orbital_groups.find(icam);
      if (it == opt.orbital_groups.end())
        vw::vw_throw(vw::ArgumentErr() 
           << "addRollYawConstraint: Failed to find orbital group for camera.\n"); 
      int group_id = it->second;

      int index_in_group = indexInGroup(icam, opt.orbital_groups);
      std::vector<double> positions = orbital_group_positions[group_id];
      std::vector<double> quaternions = orbital_group_quaternions[group_id];
      if (positions.size() / NUM_XYZ_PARAMS < 2) {
        // It can happen that we have just one frame camera, but then we just
        // can't add this constraint
        vw::vw_out(vw::WarningMessage) << "Cannot add roll and/or yaw constraint for "
          << "for an orbital group consisting of only one frame camera.\n";
        continue;
      }
        
      ceres::CostFunction* roll_yaw_cost_function
        = weightedRollYawError::Create(positions, quaternions, 
                                   georef, index_in_group,
                                   opt.roll_weight, opt.yaw_weight, 
                                   opt.initial_camera_constraint);

      // We use no loss function, as the quaternions have no outliers
      ceres::LossFunction* roll_yaw_loss_function = NULL;

      // Note how we set the quaternions to be optimized from frame_params.
      // Above, we only cared for initial positions and quaternions.
      double * curr_params = &frame_params[icam * (NUM_XYZ_PARAMS + NUM_QUAT_PARAMS)];
      problem.AddResidualBlock(roll_yaw_cost_function, roll_yaw_loss_function,
                                &curr_params[NUM_XYZ_PARAMS]); // quat starts here

      // The recorded weight should not be 0 as we will divide by it
      weight_per_residual.push_back(opt.roll_weight || 1.0);
      weight_per_residual.push_back(opt.yaw_weight  || 1.0);
    } else {
      vw::vw_throw(vw::ArgumentErr() 
         << "addRollYawConstraint: Expecting CSM linescan or frame cameras.\n");
    }

  } // end loop through cameras

  return;
}

// Apply the input adjustments to the CSM cameras. Resample linescan models.
// Get pointers to the underlying CSM cameras, as need to manipulate
// those directly. This modifies camera_models in place.
void prepareCsmCameras(Options const& opt,
  std::vector<vw::CamPtr>      const& camera_models,
  std::vector<asp::CsmModel*>       & csm_models) {

  // Wipe the output
  csm_models.clear();
  
  for (size_t icam = 0; icam < camera_models.size(); icam++) {
    asp::CsmModel * csm_cam = asp::csm_model(camera_models[icam], opt.stereo_session);

    // Sanity check
    if (csm_cam == NULL)
      vw::vw_throw(vw::ArgumentErr() << "Expecting CSM cameras.\n");

    if (!opt.input_prefix.empty()) {
      std::string adjust_file
        = asp::bundle_adjust_file_name(opt.input_prefix, opt.image_files[icam],
                                      opt.camera_files[icam]);
      vw_out() << "Reading input adjustment: " << adjust_file << std::endl;
      // This modifies camera_models
      vw::camera::AdjustedCameraModel
        adj_cam(vw::camera::unadjusted_model(camera_models[icam]));
      adj_cam.read(adjust_file);
      vw::Matrix4x4 ecef_transform = adj_cam.ecef_transform();
      csm_cam->applyTransform(ecef_transform);
    }

    // Get the underlying linescan model or frame model
    UsgsAstroLsSensorModel * ls_model
      = dynamic_cast<UsgsAstroLsSensorModel*>((csm_cam->m_gm_model).get());
    UsgsAstroFrameSensorModel * frame_model
      = dynamic_cast<UsgsAstroFrameSensorModel*>((csm_cam->m_gm_model).get());

    if (ls_model == NULL && frame_model == NULL)
      vw_throw(ArgumentErr() << "Expecting the cameras to be of CSM linescan or frame type.\n");

    // Normalize quaternions. Later, the quaternions being optimized will
    // be kept close to being normalized.  This makes it easy to ensure
    // that quaternion interpolation gives good results, especially that
    // some quaternions may get optimized and some not.
    if (ls_model != NULL) {
      asp::normalizeQuaternions(ls_model);
      // The provided tabulated positions, velocities and quaternions may be too few,
      // so resample them with --num-lines-per-position and --num-lines-per-orientation,
      // if those are set.
      resampleModel(opt, ls_model);
    } else if (frame_model != NULL) {
      normalizeQuaternions(frame_model);
    } else {
      vw::vw_throw(vw::ArgumentErr() 
        << "Expecting the cameras to be of CSM linescan or frame type.\n");
    }

    csm_models.push_back(csm_cam);
  }
}

// Create structures for pixels, xyz, and weights, to be used in optimization
// TODO(oalexan1): Avoid using xyz_vec_ptr. If the vector gets resized, the pointers
// will be invalidated.
typedef vw::ba::CameraRelationNetwork<vw::ba::JFeature> CrnT;
typedef boost::shared_ptr<Vector3> Vec3Ptr;
void createProblemStructure(Options                      const& opt,
                            CrnT                         const& crn,
                            vw::ba::ControlNetwork       const& cnet, 
                            std::set<int>                const& outliers,
                            // Outputs
                            std::vector<double>               & tri_points_vec,
                            std::vector<std::vector<Vector2>> & pixel_vec,
                            std::vector<std::vector<Vec3Ptr>> & xyz_vec,
                            std::vector<std::vector<double*>> & xyz_vec_ptr,
                            std::vector<std::vector<double>>  & weight_vec,
                            std::vector<std::vector<int>>     & isAnchor_vec) {

  int num_cameras = opt.camera_models.size();

  pixel_vec.resize(num_cameras);
  xyz_vec.resize(num_cameras);
  xyz_vec_ptr.resize(num_cameras);
  weight_vec.resize(num_cameras);
  isAnchor_vec.resize(num_cameras);

  for (int icam = 0; icam < (int)crn.size(); icam++) {
    for (auto fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++) {
      
      // The index of the 3D point
      int ipt = (**fiter).m_point_id;
      
      if (outliers.find(ipt) != outliers.end())
        continue; // Skip outliers
      
      // The observed value for the projection of point with index ipt into
      // the camera with index icam.
      Vector2 observation = (**fiter).m_location;

      // Ideally this point projects back to the pixel observation.
      double * tri_point = &tri_points_vec[0] + ipt * NUM_XYZ_PARAMS;

      double weight = 1.0;
      
      pixel_vec[icam].push_back(observation);
      weight_vec[icam].push_back(weight);
      isAnchor_vec[icam].push_back(0);
      // It is bad logic to store pointers as below. What if tri_points_vec
      // later gets resized?
      xyz_vec_ptr[icam].push_back(tri_point); 
    }
  }

  return;
}

// Put the triangulated points in a vector. Update the cnet from the DEM,
// if we have one.
void formTriVec(std::vector<Vector3> const& dem_xyz_vec,
                bool have_dem,
                // Outputs
                ba::ControlNetwork  & cnet,
                std::vector<double> & tri_points_vec) {

  int num_tri_points = cnet.size();
  if (num_tri_points == 0)
    vw::vw_throw(ArgumentErr() << "No triangulated ground points were found.\n"); 

  tri_points_vec.resize(num_tri_points*NUM_XYZ_PARAMS, 0.0);

  for (int ipt = 0; ipt < num_tri_points; ipt++) {
    // We overwrite the triangulated point when we have an input DEM.
    // It is instructive to examine the pointmap residual file to see
    // what effect that has on residuals.  This point will likely try
    // to move back somewhat to its triangulated position during
    // optimization, depending on the strength of the weight which
    // tries to keep it back in place.
    Vector3 tri_point = cnet[ipt].position();
    if (have_dem && dem_xyz_vec.at(ipt) != Vector3(0, 0, 0)) {
      tri_point = dem_xyz_vec.at(ipt);

      // Update in the cnet too
      cnet[ipt].set_position(Vector3(tri_point[0], tri_point[1], tri_point[2]));
      
      // Ensure we can track it later
      cnet[ipt].set_type(vw::ba::ControlPoint::PointFromDem); 
    }
    
    for (int q = 0; q < NUM_XYZ_PARAMS; q++)
      tri_points_vec[ipt*NUM_XYZ_PARAMS + q] = tri_point[q];
  }
  return;
}

void run_jitter_solve(int argc, char* argv[]) {

  // Parse arguments and perform validation
  Options opt;
  handle_arguments(argc, argv, opt);

  bool approximate_pinhole_intrinsics = false;
  asp::load_cameras(opt.image_files, opt.camera_files, opt.out_prefix, opt,  
                    approximate_pinhole_intrinsics,  
                    // Outputs
                    opt.stereo_session,  // may change
                    opt.single_threaded_cameras,  
                    opt.camera_models);

  // Find the datum
  vw::cartography::Datum datum;
  asp::datum_from_cameras(opt.image_files, opt.camera_files,  
                          opt.stereo_session,  // may change
                          // Outputs
                          datum);
  
  // Apply the input adjustments to the cameras. Resample linescan models.
  // Get pointers to the underlying CSM cameras, as need to manipulate
  // those directly.
  std::vector<asp::CsmModel*> csm_models;
  prepareCsmCameras(opt, opt.camera_models, csm_models);
  
  // Make a list of all the image pairs to find matches for. Some quantities
  // below are not needed but are part of the API.
  bool got_est_cam_positions = false;
  double position_filter_dist = -1.0;
  std::vector<vw::Vector3> estimated_camera_gcc;
  bool have_overlap_list = false;
  std::set<std::pair<std::string, std::string>> overlap_list;
  std::vector<std::pair<int,int>> all_pairs;
  asp::determine_image_pairs(// Inputs
                             opt.overlap_limit, opt.match_first_to_last,  
                             opt.image_files, 
                             got_est_cam_positions, position_filter_dist,
                             estimated_camera_gcc, have_overlap_list, overlap_list,
                             // Output
                             all_pairs);

  // List existing match files. This can take a while.
  vw_out() << "Computing the list of existing match files.\n";
  std::string prefix = asp::match_file_prefix(opt.clean_match_files_prefix,
                                              opt.match_files_prefix,  
                                              opt.out_prefix);
  std::set<std::string> existing_files;
  asp::listExistingMatchFiles(prefix, existing_files);

  // Load match files
  std::map<std::pair<int, int>, std::string> match_files;
  for (size_t k = 0; k < all_pairs.size(); k++) {
    int i = all_pairs[k].first;
    int j = all_pairs[k].second;
    std::string const& image1_path  = opt.image_files[i];  // alias
    std::string const& image2_path  = opt.image_files[j];  // alias
    std::string const& camera1_path = opt.camera_files[i]; // alias
    std::string const& camera2_path = opt.camera_files[j]; // alias

      // Load match files from a different source
    std::string match_file 
      = asp::match_filename(opt.clean_match_files_prefix, opt.match_files_prefix,  
                            opt.out_prefix, image1_path, image2_path);

    // The external match file does not exist, don't try to load it
    if (existing_files.find(match_file) == existing_files.end())
      continue;
    
    match_files[std::make_pair(i, j)] = match_file;
  }

  // Build control network and perform triangulation with adjusted input cameras
  ba::ControlNetwork cnet("jitter_solve");
  bool triangulate_control_points = true;
  bool success = vw::ba::build_control_network(triangulate_control_points,
                                               cnet, // output
                                               opt.camera_models, opt.image_files,
                                               match_files, opt.min_matches,
                                               opt.min_triangulation_angle*(M_PI/180.0),
                                               opt.forced_triangulation_distance,
                                               opt.max_pairwise_matches);
  if (!success)
    vw::vw_throw(vw::ArgumentErr()
             << "Failed to build a control network. Check the bundle adjustment directory "
             << "for clean matches. Or, consider removing all .vwip and "
             << ".match files and increasing the number of interest points "
             << "using --ip-per-image or --ip-per-tile, or decreasing --min-matches, "
             << "and then re-running bundle adjustment.\n");

  // TODO(oalexan1): Is it possible to avoid using CRNs?
  vw::ba::CameraRelationNetwork<vw::ba::JFeature> crn;
  crn.from_cnet(cnet);
  
  if ((int)crn.size() != opt.camera_models.size()) 
    vw_throw(ArgumentErr() << "Book-keeping error, the size of CameraRelationNetwork "
             << "must equal the number of images.\n");

  // Flag as outliers points with initial reprojection error bigger than
  // a certain amount. This assumes that the input cameras are very accurate.
  std::set<int> outliers;
  flag_initial_outliers(cnet, crn, opt.camera_models, opt.max_init_reproj_error,  
                        // Output
                        outliers);
  vw_out() << "Removed " << outliers.size() 
    << " outliers based on initial reprojection error.\n";
  
  bool have_dem = (!opt.heights_from_dem.empty() || !opt.ref_dem.empty());

  // Create anchor xyz with the help of a DEM in two ways.
  // TODO(oalexan1): Study how to best pass the DEM to avoid the code
  // below not being slow. It is not clear if the DEM tiles are cached
  // when passing around an ImageViewRef.
  std::vector<Vector3> dem_xyz_vec;
  vw::cartography::GeoReference dem_georef, anchor_georef;
  ImageViewRef<PixelMask<double>> interp_dem, interp_anchor_dem;
  if (opt.heights_from_dem != "") {
    asp::create_interp_dem(opt.heights_from_dem, dem_georef, interp_dem);
    asp::update_point_height_from_dem(cnet, outliers, dem_georef, interp_dem,  
                                      // Output
                                      dem_xyz_vec);
  } else if (opt.ref_dem != "") {
    asp::create_interp_dem(opt.ref_dem, dem_georef, interp_dem);
    asp::calc_avg_intersection_with_dem(cnet, crn, outliers, opt.camera_models,
                                        dem_georef, interp_dem,
                                        // Output
                                        dem_xyz_vec);
  }
  
  if (opt.anchor_dem != "")
    asp::create_interp_dem(opt.anchor_dem, anchor_georef, interp_anchor_dem);

  // Handle the roll/yaw constraint DEM. We already checked that one of thse cases should work
  vw::cartography::GeoReference roll_yaw_georef;
  if (opt.roll_weight > 0 || opt.yaw_weight > 0) {
    if (opt.heights_from_dem != "" || opt.ref_dem != "") {
      roll_yaw_georef = dem_georef;
      vw::vw_out() << "Using the DEM from --heights-from-dem or --reference-dem "
                   << "for the roll/yaw constraint.\n";
    } else if (opt.anchor_dem != "") {
      roll_yaw_georef = anchor_georef;
      vw::vw_out() << "Using the DEM from --anchor-dem for the roll/yaw constraint.\n";
    }
  } 

  int num_cameras = opt.camera_models.size();
  if (num_cameras < 2)
    vw_throw(ArgumentErr() << "Expecting at least two input cameras.\n");

  // If some of the input cameras are frame, need to store position and
  // quaternion variables for them outside the camera model, as these are
  // private for UsgsAstroFrameCameraModel, unlike for UsgsAstroLsSensorModel.
  // It is easier to just allocate the space for all cameras, even if it may go
  // unused mostly or at all.
  std::vector<double> frame_params;
  initFrameCameraParams(csm_models, frame_params);

  // Put the triangulated points in a vector. Update the cnet from the DEM,
  // if we have one.
  std::vector<double> tri_points_vec;
  formTriVec(dem_xyz_vec, have_dem, 
    cnet, tri_points_vec); // outputs
  
  // Create structures for pixels, xyz, and weights, to be used in optimization
  std::vector<std::vector<Vector2>> pixel_vec;
  std::vector<std::vector<boost::shared_ptr<Vector3>>> xyz_vec;
  std::vector<std::vector<double*>> xyz_vec_ptr;
  std::vector<std::vector<double>> weight_vec;
  std::vector<std::vector<int>> isAnchor_vec;
  createProblemStructure(opt, crn, cnet, outliers,
                         // Outputs
                         tri_points_vec, pixel_vec, xyz_vec, xyz_vec_ptr,
                         weight_vec, isAnchor_vec);

  // Find anchor points and append to pixel_vec, weight_vec, etc.
  if (opt.num_anchor_points > 0 && opt.anchor_weight > 0)
    calcAnchorPoints(opt, interp_anchor_dem, anchor_georef, csm_models,  
                     // Append to these
                     pixel_vec, xyz_vec, xyz_vec_ptr, weight_vec, isAnchor_vec);

  // Need this in order to undo the multiplication by weight before saving the residuals
  std::vector<double> weight_per_residual;

  // The problem to solve
  ceres::Problem problem;
  
  // Add reprojection errors
  addReprojectionErrors(opt, crn, pixel_vec, xyz_vec, xyz_vec_ptr, weight_vec,
                        isAnchor_vec, csm_models,
                        // Outputs
                        frame_params, weight_per_residual, problem);
 
  // Add the DEM constraint. We check earlier that only one
  // of the two options below can be set at a time.
  if (have_dem)
    addDemConstraint(opt, xyz_vec, xyz_vec_ptr, dem_xyz_vec, outliers, cnet,  
                     // Outputs
                     tri_points_vec, 
                     weight_per_residual,  // append
                     problem);

  // Add the constraint to keep triangulated points close to initial values
  // This does not need a DEM or alignment.
  // This must happen after any DEM-based constraint is set, and won't
  // apply to tri points already constrained by the DEM (so it will
  // work only where the DEM is missing).
  if (opt.tri_weight > 0) 
    addTriConstraint(opt, outliers, cnet,  
                     // Outputs
                     tri_points_vec,  
                     weight_per_residual,  // append
                     problem);

  // Add constraints to keep quat norm close to 1, and make rotations and translations
  // not change too much
  addQuatNormRotationTranslationConstraints(opt, outliers, crn, csm_models,  
                                            // Outputs
                                            frame_params,
                                            tri_points_vec,  
                                            weight_per_residual,  // append
                                            problem);

  if (opt.roll_weight > 0 || opt.yaw_weight > 0)
    addRollYawConstraint(opt, crn, csm_models, roll_yaw_georef,
                        frame_params, weight_per_residual, problem); // outputs

  // Save residuals before optimization
  std::string residual_prefix = opt.out_prefix + "-initial_residuals";
  save_residuals(residual_prefix, problem, opt, cnet, crn, have_dem, datum,
                 tri_points_vec, dem_xyz_vec, outliers, weight_per_residual,
                 // These are needed for anchor points
                 pixel_vec, xyz_vec, xyz_vec_ptr, weight_vec, isAnchor_vec);
  
  // Set up the problem
  ceres::Solver::Options options;
  options.gradient_tolerance  = 1e-16;
  options.function_tolerance  = 1e-16;
  options.parameter_tolerance = opt.parameter_tolerance; // default is 1e-12
  options.max_num_iterations                = opt.num_iterations;
  options.max_num_consecutive_invalid_steps = std::max(20, opt.num_iterations/5); // try hard
  options.minimizer_progress_to_stdout      = true;
  if (opt.single_threaded_cameras)
    options.num_threads = 1;
  else
    options.num_threads = opt.num_threads;
  // This is supposed to help with speed in a certain size range
  options.linear_solver_type = ceres::SPARSE_SCHUR;
  options.use_explicit_schur_complement = true; 
  options.linear_solver_type  = ceres::ITERATIVE_SCHUR;
  options.preconditioner_type = ceres::SCHUR_JACOBI;
  options.use_explicit_schur_complement = false; // Only matters with ITERATIVE_SCHUR
  
  // Solve the problem
  vw_out() << "Starting the Ceres optimizer." << std::endl;
  ceres::Solver::Summary summary;
  ceres::Solve(options, &problem, &summary);
  vw_out() << summary.FullReport() << "\n";
  if (summary.termination_type == ceres::NO_CONVERGENCE) 
    vw_out() << "Found a valid solution, but did not reach the actual minimum.\n";

  // With the problem solved, update camera_models based on frame_params
  // (applies only to frame cameras, if any)
  updateFrameCameras(csm_models, frame_params);  

  // Save residuals after optimization
  // TODO(oalexan1): Add here the anchor residuals
  residual_prefix = opt.out_prefix + "-final_residuals";
  save_residuals(residual_prefix, problem, opt, cnet, crn, have_dem, datum,
                 tri_points_vec, dem_xyz_vec, outliers, weight_per_residual,
                 // These are needed for anchor points
                 pixel_vec, xyz_vec, xyz_vec_ptr, weight_vec, isAnchor_vec);

  // TODO(oalexan1): Make this a function
  // Save the optimized model states. Note that we optimized directly the camera
  // model states, so there's no need to update them from some optimization
  // workspace.
  for (size_t icam = 0; icam < opt.camera_models.size(); icam++) {
    std::string adjustFile = asp::bundle_adjust_file_name(opt.out_prefix,
                                                          opt.image_files[icam],
                                                          opt.camera_files[icam]);
    std::string csmFile = asp::csmStateFile(adjustFile);
    asp::CsmModel * csm_cam = asp::csm_model(opt.camera_models[icam], opt.stereo_session);
    csm_cam->saveState(csmFile);
  }
  
  return;
}

} // end namespace asp
  
int main(int argc, char* argv[]) {

  try {
    xercesc::XMLPlatformUtils::Initialize();
    
    asp::run_jitter_solve(argc, argv);
    
    xercesc::XMLPlatformUtils::Terminate();
  } ASP_STANDARD_CATCHES;

  return 0;
}
