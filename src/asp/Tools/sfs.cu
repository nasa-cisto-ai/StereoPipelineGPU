#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// TODO(oalexan1): Move image logic to SfsImageProc.cc. Add also
// SfS camera approx logic, and SfS cost function logic.
// Remove all logic with multiple DEM clips, it turned out not to work.
// Remove all floating of cameras from the code and the doc, one has to use bundle adjust.
// Then also remove all the logic using unadjusted cameras except the place
// when the sun positions are read.
// Evaluate RPC accuracy.
// Deal with outliers in image intensity.
// The rpc approximation should also approximate the adjusted cameras if those won't float.
// This approximation needs to remember is domain of validity.
// TODO: Ensure the output DEM is float. Check its no-data value.
// TODO: Study more floating model coefficients.
// TODO: Study why using tabulated camera model and multiple resolutions does
// not work as well as it should.
// TODO: When using approx camera, we assume the DEM and image grids are very similar.
// TODO: Remove warning from the approx camera
// TODO: Make it possible to initialize a DEM from scratch.
// TODO: Study more the multi-resolution approach.
// TODO: Must specify in the SfS doc that the Lunar-Lambertian model fails at poles
// TODO: If this code becomes multi-threaded, need to keep in mind
// that camera models are shared and modified, so
// this may cause problems.
// TODO: How to relax conditions at the boundary to improve the accuracy?
// TODO: Study if blurring the input images improves the fit.
// TODO: Add --orthoimage option, and make it clear where the final DEM is.
// Same for albedo. The other info should be printed only in debug mode.
// Implement multi-grid for SfS, it should help with bad initial DEM and bad
// camera alignment.
// TODO: Save final DEM and final ortho images.
// Say that if the camera are bundle adjusted, sfs can further improve
// the camera positions to make the results more self consistent,
// but this works only if the cameras are reasonably accurate to start with.
// TODO: Study the effect of reading the images as double as opposed to float.
// TODO: Study the effect of using bicubic interpolation.
// TODO: Study phaseCoeffC1, etc.
// TODO: Find a good automatic value for the smoothness weight.
// TODO: How to change the smoothness weight if resolution changes?
// How to change the smoothness weight if the number of images changes?
// TODO: Investigate the sign of the normal.
// TODO: Loop over all images when doing sfs.
// TODO: Check that we are within image boundaries when interpolating.
// TODO: Radiometric calibration of images.
// TODO: Add various kind of loss function.
// TODO: Study the normal computation formula.
// TODO: Move some code to Core.
// TODO: Make it work with non-ISIS cameras.
// TODO: Clean up some of the classes, not all members are needed.

/// \file sfs.cc

// Turn off warnings from boost and other packages
#if defined(__GNUC__) || defined(__GNUG__)
#define LOCAL_GCC_VERSION (__GNUC__ * 10000             \
                           + __GNUC_MINOR__ * 100       \
                           + __GNUC_PATCHLEVEL__)
#if LOCAL_GCC_VERSION >= 40600
#pragma GCC diagnostic push
#endif
#if LOCAL_GCC_VERSION >= 40202
#pragma GCC diagnostic ignored "-Wunused-local-typedefs"
#endif
#endif

#include <vw/Image/MaskViews.h>
#include <vw/Image/AntiAliasing.h>
#include <vw/Image/InpaintView.h>
#include <vw/Image/DistanceFunction.h>
#include <vw/Cartography/GeoReferenceUtils.h>
#include <vw/Core/Stopwatch.h>
#include <vw/Core/CmdUtils.h>

#include <asp/Core/Macros.h>
#include <asp/Core/Common.h>
#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/IsisIO/IsisCameraModel.h>
#include <asp/Camera/CsmModel.h>
#include <asp/Core/BundleAdjustUtils.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Core/SfsImageProc.h>
#include <asp/Camera/RPCModelGen.h>

#include <ceres/ceres.h>
#include <ceres/loss_function.h>

#include <iostream>
#include <stdexcept>
#include <string>
#include<sys/types.h>

#if defined(__GNUC__) || defined(__GNUG__)
#if LOCAL_GCC_VERSION >= 40600
#pragma GCC diagnostic pop
#endif
#undef LOCAL_GCC_VERSION
#endif

namespace po = boost::program_options;
namespace fs = boost::filesystem;

int g_num_locks = 0;
int g_warning_count = 0;
int g_max_warning_count = 1000;
const size_t g_num_model_coeffs = 16;
const size_t g_max_num_haze_coeffs = 6; // see nonlin_reflectance()

// If the blend weight is ground weight, rather than image weight,
// use it as it is, without projecting into camera and interpolating.
// It is a lot of work to pass this all over the place.
bool g_blend_weight_is_ground_weight = false;

using namespace vw;
using namespace vw::camera;
using namespace vw::cartography;

typedef ImageViewRef<PixelMask<float>> MaskedImgT;
typedef ImageViewRef<double> DoubleImgT;

namespace vw { namespace camera {

  // A base approx camera model class that will factor out some functionality
  // from the two approx camera model classes we have below.
  class ApproxBaseCameraModel: public CameraModel {

  protected:
    BBox2i m_img_bbox;
    mutable BBox2 m_point_box, m_crop_box;
    bool m_model_is_valid;
    boost::shared_ptr<CameraModel> m_exact_unadjusted_camera;
    AdjustedCameraModel m_exact_adjusted_camera;

  public:

    ApproxBaseCameraModel(AdjustedCameraModel const& exact_adjusted_camera,
                          boost::shared_ptr<CameraModel> exact_unadjusted_camera,
                          BBox2i img_bbox): m_exact_adjusted_camera(exact_adjusted_camera),
                                            m_exact_unadjusted_camera(exact_unadjusted_camera),
                                            m_img_bbox(img_bbox){}
    
    // The range of pixels in the image we are actually expected to use.
    // Note that the function returns an alias, so that we can modify the
    // crop box from outside.
    BBox2 & crop_box(){
      m_crop_box.crop(m_img_bbox);
      return m_crop_box;
    }

    bool model_is_valid(){
      return m_model_is_valid;
    }
    
    boost::shared_ptr<CameraModel> exact_unadjusted_camera() const{
      return m_exact_unadjusted_camera;
    }
    
    AdjustedCameraModel exact_adjusted_camera() const{
      return m_exact_adjusted_camera;
    }
    
  };
    
  // This class provides an approximation for the point_to_pixel()
  // function of an ISIS camera around a current DEM. The algorithm
  // works by tabulation of point_to_pixel and pixel_to_vector values
  // at the mean dem height.
  class ApproxCameraModel: public ApproxBaseCameraModel {
    mutable Vector3 m_mean_dir; // mean vector from camera to ground
    GeoReference m_geo;
    double m_mean_ht;
    mutable ImageView< PixelMask<Vector3> > m_pixel_to_vec_mat;
    mutable ImageView< PixelMask<Vector2> > m_point_to_pix_mat;
    double m_approx_table_gridx, m_approx_table_gridy;
    bool m_use_rpc_approximation, m_use_semi_approx;
    vw::Mutex& m_camera_mutex;
    Vector2 m_uncompValue;
    mutable int m_begX, m_endX, m_begY, m_endY;
    mutable bool m_compute_mean, m_stop_growing_range;
    mutable int m_count;
    boost::shared_ptr<asp::RPCModel> m_rpc_model;
    
    bool comp_rpc_approx_table(AdjustedCameraModel const& adj_camera,
                               boost::shared_ptr<CameraModel> exact_unadjusted_camera,
                               BBox2i img_bbox,
                               ImageView<double> const& dem,
                               GeoReference const& geo,
                               double rpc_penalty_weight){

      try {
        // Generate point pairs
        std::vector<Vector3> all_llh;
        std::vector<Vector2> all_pixels;

        vw_out() << "Projecting pixels into the camera to generate the RPC model.\n";
        vw::TerminalProgressCallback tpc("asp", "\t--> ");
        double inc_amount = 1.0 / double(dem.cols());
        tpc.report_progress(0);

        // If the DEM is too big, we need to skip points. About
        // 40,000 points should be good enough to determine 78 RPC
        // coefficients.
        double num = 200.0;
        double delta_col = std::max(1.0, dem.cols()/double(num));
        double delta_row = std::max(1.0, dem.rows()/double(num));
        BBox3 llh_box;
        BBox2 pixel_box;
        for (double dcol = 0; dcol < dem.cols(); dcol += delta_col) {
          for (double drow = 0; drow < dem.rows(); drow += delta_row) {
            int col = dcol, row = drow; // cast to int
            Vector2 pix(col, row);
            Vector2 lonlat = geo.pixel_to_lonlat(pix);
          
            // Lon lat height
            Vector3 llh;
            llh[0] = lonlat[0]; llh[1] = lonlat[1]; llh[2] = dem(col, row);
            Vector3 xyz = geo.datum().geodetic_to_cartesian(llh);

            // Go back to llh. This is a bugfix for the 360 deg offset problem.
            llh = geo.datum().cartesian_to_geodetic(xyz);
          
            Vector2 cam_pix = exact_unadjusted_camera->point_to_pixel(xyz);
            //if (!m_img_bbox.contains(cam_pix)) 
            //  continue; // skip out of range pixels? Not a good idea.
         
            if (m_img_bbox.contains(cam_pix)) 
              m_crop_box.grow(cam_pix);

            all_llh.push_back(llh);
            all_pixels.push_back(cam_pix);

            llh_box.grow(llh);
            pixel_box.grow(cam_pix);
          }

          tpc.report_incremental_progress( inc_amount );
        }
        tpc.report_finished();
      
        BBox2 ll_box;
        ll_box.min() = subvector(llh_box.min(), 0, 2);
        ll_box.max() = subvector(llh_box.max(), 0, 2);

        BBox2 cropped_pixel_box = pixel_box;
        cropped_pixel_box.crop(m_img_bbox);
        if (cropped_pixel_box.empty()) {
          vw_out() << "No points fall into the camera.\n";
          return false;
        }

        if (ll_box.empty()) {
          vw_out() << "Empty lon-lat box.\n";
          return false;
        }

        // This is a bugfix. The RPC approximation works best when the
        // input llh points are in an llh box whose sides are vertical
        // and horizontal, rather than in a box which is rotated.
        vw_out() << "Re-projecting pixels into the camera to improve accuracy.\n";
        vw::TerminalProgressCallback tpc2("asp", "\t--> ");
        double inc_amount2 = 1.0 / double(num);
        tpc2.report_progress(0);
        llh_box = BBox3();
        pixel_box = BBox2();
        all_llh.clear();
        all_pixels.clear();
        ImageViewRef<double> interp_dem
          = interpolate(dem, BicubicInterpolation(), ConstantEdgeExtension());
        double delta_lon = (ll_box.max()[0] - ll_box.min()[0])/double(num);
        double delta_lat = (ll_box.max()[1] - ll_box.min()[1])/double(num);
        for (double lon = ll_box.min()[0]; lon <= ll_box.max()[0] + delta_lon; lon += delta_lon) {
          for (double lat = ll_box.min()[1]; lat <= ll_box.max()[1] + delta_lat; lat += delta_lat) {

            Vector2 pix = geo.lonlat_to_pixel(Vector2(lon, lat));
            if (pix[0] < 0 || pix[0] > dem.cols()-1) continue;
            if (pix[1] < 0 || pix[1] > dem.rows()-1) continue;
            double ht = interp_dem(pix[0], pix[1]);

            // Lon lat height
            Vector3 llh;
            llh[0] = lon; llh[1] = lat; llh[2] = ht;
            Vector3 xyz = geo.datum().geodetic_to_cartesian(llh);

            // Later we will project DEM points into the adjusted camera.
            // That is the same as projecting adjusted points into the exact camera.
            // Hence, develop the RPC approximation using adjusted points.
            // TODO: Maybe we should also ensure that the unadjusted xyz is
            // also part of the model building? But probably not, as usually
            // adjustments change very little, and this will increase run-time by 2x.
            xyz = adj_camera.adjusted_point(xyz);
            
            // Go back to llh. This is a bugfix for the 360 deg offset problem.
            llh = geo.datum().cartesian_to_geodetic(xyz);

            Vector2 cam_pix = exact_unadjusted_camera->point_to_pixel(xyz);
            //if (!m_img_bbox.contains(cam_pix)) 
            //  continue; // skip out of range pixels? Not a good idea.
         
            if (m_img_bbox.contains(cam_pix)) 
              m_crop_box.grow(cam_pix);

            all_llh.push_back(llh);
            all_pixels.push_back(cam_pix);
            
            llh_box.grow(llh);
            pixel_box.grow(cam_pix);
          }
          
          tpc2.report_incremental_progress( inc_amount2 );
        }
        tpc2.report_finished();

        cropped_pixel_box = pixel_box;
        cropped_pixel_box.crop(m_img_bbox);
        if (cropped_pixel_box.empty()) {
          vw_out() << "No points fall into the camera.\n";
          return false;
        }

        ll_box.min() = subvector(llh_box.min(), 0, 2);
        ll_box.max() = subvector(llh_box.max(), 0, 2);
        if (ll_box.empty()) {
          vw_out() << "Empty lon-lat box.\n";
          return false;
        }
        
        Vector3 llh_scale  = (llh_box.max() - llh_box.min())/2.0; // half range
        Vector3 llh_offset = (llh_box.max() + llh_box.min())/2.0; // center point
      
        Vector2 pixel_scale  = (pixel_box.max() - pixel_box.min())/2.0; // half range 
        Vector2 pixel_offset = (pixel_box.max() + pixel_box.min())/2.0; // center point

        // Ensure we never divide by zero. For example, if the input dem heights are all constant,
        // then the height scale will be zero from above.
        for (size_t i = 0; i < llh_scale.size(); i++) 
          if (llh_scale[i] == 0) llh_scale[i] = 1;
        for (size_t i = 0; i < pixel_scale.size(); i++) 
          if (pixel_scale[i] == 0) pixel_scale[i] = 1;
        
        vw_out() << "Lon-lat-height box for the RPC approx: " << llh_box   << std::endl;
        vw_out() << "Camera pixel box for the RPC approx:   " << pixel_box << std::endl;

        Vector<double> normalized_llh;
        Vector<double> normalized_pixels;
        int num_total_pts = all_llh.size();
        normalized_llh.set_size(asp::RPCModel::GEODETIC_COORD_SIZE*num_total_pts);
        normalized_pixels.set_size(asp::RPCModel::IMAGE_COORD_SIZE*num_total_pts
                                   + asp::RpcSolveLMA::NUM_PENALTY_TERMS);
        for (size_t i = 0; i < normalized_pixels.size(); i++) {
          // Important: The extra penalty terms are all set to zero here.
          normalized_pixels[i] = 0.0; 
        }
      
        // Form the arrays of normalized pixels and normalized llh
        for (int pt = 0; pt < num_total_pts; pt++) {
    
          // Normalize the pixel to -1 <> 1 range
          Vector3 llh_n   = elem_quot(all_llh[pt]    - llh_offset,   llh_scale);
          Vector2 pixel_n = elem_quot(all_pixels[pt] - pixel_offset, pixel_scale);
          subvector(normalized_llh, asp::RPCModel::GEODETIC_COORD_SIZE*pt,
                    asp::RPCModel::GEODETIC_COORD_SIZE) = llh_n;
          subvector(normalized_pixels, asp::RPCModel::IMAGE_COORD_SIZE*pt,
                    asp::RPCModel::IMAGE_COORD_SIZE   ) = pixel_n;
    
        }

        // Find the RPC coefficients
        asp::RPCModel::CoeffVec line_num, line_den, samp_num, samp_den;
        std::string output_prefix = "";
        vw_out() << "Generating the RPC approximation using " << num_total_pts << " point pairs.\n";
        asp::gen_rpc(// Inputs
                     rpc_penalty_weight, output_prefix,
                     normalized_llh, normalized_pixels,  
                     llh_scale, llh_offset, pixel_scale, pixel_offset,
                     // Outputs
                     line_num, line_den, samp_num, samp_den);
      
        m_rpc_model = boost::shared_ptr<asp::RPCModel>
          (new asp::RPCModel(geo.datum(), line_num, line_den,
                             samp_num, samp_den, pixel_offset, pixel_scale,
                             llh_offset, llh_scale));
      } catch (std::exception const& e) {
        vw_out() << e.what() << std::endl;
        return false;
      }      
      
      return true;
    }
    
    void comp_entries_in_table() const{
      for (int x = m_begX; x <= m_endX; x++) {
        for (int y = m_begY; y <= m_endY; y++) {
      
          // This will be useful when we invoke this function repeatedly
          if (m_point_to_pix_mat(x, y).child() != m_uncompValue) {
            continue;
          }
      
          Vector2 pt(m_point_box.min().x() + x*m_approx_table_gridx,
                     m_point_box.min().y() + y*m_approx_table_gridy);
          Vector2 lonlat = m_geo.point_to_lonlat(pt);
          Vector3 xyz = m_geo.datum().geodetic_to_cartesian
            (Vector3(lonlat[0], lonlat[1], m_mean_ht));
          bool success = true;
          Vector2 pix;
          Vector3 vec;
          try {
            pix = m_exact_unadjusted_camera->point_to_pixel(xyz);
            //if (true || m_img_bbox.contains(pix))  // Need to think more here
            vec = m_exact_unadjusted_camera->pixel_to_vector(pix);
            //else
            // success = false;
            
          }catch(...){
            success = false;
          }
          if (success) {
            m_pixel_to_vec_mat(x, y) = vec;
            m_point_to_pix_mat(x, y) = pix;
            m_pixel_to_vec_mat(x, y).validate();
            m_point_to_pix_mat(x, y).validate();
            if (m_compute_mean) {
              m_mean_dir += vec; // only when the point projects inside the camera?
              if (m_img_bbox.contains(pix)) 
                m_crop_box.grow(pix);
              m_count++;
            }
          }else{
            m_pixel_to_vec_mat(x, y).invalidate();
            m_point_to_pix_mat(x, y).invalidate();
          }
        }
      }
      
    }
    
  public:

    ApproxCameraModel(AdjustedCameraModel const& exact_adjusted_camera,
                      boost::shared_ptr<CameraModel> exact_unadjusted_camera,
                      BBox2i img_bbox, 
                      ImageView<double> const& dem,
                      GeoReference const& geo,
                      double nodata_val,
                      bool use_rpc_approximation, bool use_semi_approx,
                      double rpc_penalty_weight,
                      vw::Mutex &camera_mutex):
      ApproxBaseCameraModel(exact_adjusted_camera, exact_unadjusted_camera, img_bbox),
      m_geo(geo),
      m_use_rpc_approximation(use_rpc_approximation),
      m_use_semi_approx(use_semi_approx),
      m_camera_mutex(camera_mutex) {

      // Initialize members of the base class
      m_model_is_valid = true;
      
      int big = 1e+8;
      m_uncompValue = Vector2(-big, -big);
      m_compute_mean = true; // We'll set this to false when we finish estimating the mean
      m_stop_growing_range = false; // stop when it does not help
      
      if (dynamic_cast<IsisCameraModel*>(exact_unadjusted_camera.get()) == NULL)
        vw_throw( ArgumentErr()
                  << "ApproxCameraModel: Expecting an unadjusted camera model.\n");

      // Compute the mean DEM height.
      // We expect all DEM entries to be valid.
      m_mean_ht = 0;
      double num = 0.0;
      for (int col = 0; col < dem.cols(); col++) {
        for (int row = 0; row < dem.rows(); row++) {
          if (dem(col, row) == nodata_val)
            vw_throw( ArgumentErr()
                      << "ApproxCameraModel: Expecting a DEM without nodata values.\n");
          m_mean_ht += dem(col, row);
          num += 1.0;
        }
      }
      if (num > 0) m_mean_ht /= num;

      // The area we're supposed to work around
      m_point_box = m_geo.pixel_to_point_bbox(bounding_box(dem));
      double wx = m_point_box.width(), wy = m_point_box.height();
      m_approx_table_gridx = wx/std::max(dem.cols(), 1);
      m_approx_table_gridy = wy/std::max(dem.rows(), 1);

      if (m_approx_table_gridx == 0 || m_approx_table_gridy == 0) {
        vw_throw( ArgumentErr()
                  << "ApproxCameraModel: Expecting a positive grid size.\n");
      }

      // Expand the box, as later the DEM will change. 
      double extra = 1.00; // may need to lower here!
      m_point_box.min().x() -= extra*wx; m_point_box.max().x() += extra*wx;
      m_point_box.min().y() -= extra*wy; m_point_box.max().y() += extra*wy;
      wx = m_point_box.width();
      wy = m_point_box.height();

      vw_out() << "Approximation proj box: " << m_point_box << std::endl;

      if (m_use_semi_approx)
        return;
      
      // Bypass everything if doing RPC
      if (m_use_rpc_approximation) {
        m_model_is_valid = comp_rpc_approx_table(exact_adjusted_camera,
                                                 exact_unadjusted_camera, m_img_bbox,
                                                 dem,  geo, rpc_penalty_weight);
        
        // Ensure the box is valid
        //if (m_crop_box.empty()) m_crop_box = BBox2(0, 0, 2, 2);
    
#if 1
        // Expand the box a bit, as later the DEM will change and values at some
        // new pixels will be needed.
        m_crop_box.crop(m_img_bbox);
        if (!m_crop_box.empty()) {
          double wd = m_crop_box.width();
          double ht = m_crop_box.height();
          m_crop_box.min().x() -= extra*wd; m_crop_box.max().x() += extra*wd;
          m_crop_box.min().y() -= extra*ht; m_crop_box.max().y() += extra*ht;
          m_crop_box = grow_bbox_to_int(m_crop_box);
        }
        m_crop_box.crop(m_img_bbox);
#endif

        return;
      }
      
      // We will tabulate the point_to_pixel function at a multiple of
      // the grid, and we'll use interpolation for anything in
      // between.
      //m_approx_table_gridx /= 2.0; m_approx_table_gridy /= 2.0; // fine
      m_approx_table_gridx *= 2.0; m_approx_table_gridy *= 2.0; // coarse. good enough.

      int numx = wx/m_approx_table_gridx;
      int numy = wy/m_approx_table_gridy;

      vw_out() << "Lookup table dimensions: " << numx << ' ' << numy << std::endl;

      // Choose f so that the width from m_begX to m_endX is 2 x original wx
      double f = 0; // (extra-0.5)/(2.0*extra+1.0);
      m_begX = f*numx; m_endX = std::min((1.0-f)*numx, numx-1.0);
      m_begY = f*numy; m_endY = std::min((1.0-f)*numy, numy-1.0);
      
      //vw_out() << "Size of actually pre-computed table: "
      //           << m_endX - m_begX << ' ' << m_endY - m_begY << std::endl;
      
      // Mark all values as uncomputed and invalid
      m_pixel_to_vec_mat.set_size(numx, numy);
      m_point_to_pix_mat.set_size(numx, numy);
      for (int x = 0; x < numx; x++) {
        for (int y = 0; y < numy; y++) {
          m_point_to_pix_mat(x, y) = m_uncompValue;
          m_point_to_pix_mat(x, y).invalidate();
        }
      }
      
      // Fill in the table. Find along the way the mean direction from
      // the camera to the ground. Invalid values will be masked.
      m_count = 0;
      m_mean_dir = Vector3();
      comp_entries_in_table();
      m_mean_dir /= std::max(1, m_count);
      m_mean_dir = m_mean_dir/norm_2(m_mean_dir);
      m_compute_mean = false; // done computing the mean
      
      // Ensure the box is valid
      //if (m_crop_box.empty()) m_crop_box = BBox2(0, 0, 2, 2);

#if 1
      // Expansion should not be necessary, as we already expanded
      // m_point_box and we used that expanded box to compute m_crop_box.
      m_crop_box.crop(m_img_bbox);
      if (!m_crop_box.empty()) {
        // Expand the box a bit, as later the DEM will change and values at some
        // new pixels will be needed.
        double wd = m_crop_box.width();
        double ht = m_crop_box.height();
        double extra2 = 0.25; // still, just in case, a bit more expansion
        m_crop_box.min().x() -= extra2*wd; m_crop_box.max().x() += extra2*wd;
        m_crop_box.min().y() -= extra2*ht; m_crop_box.max().y() += extra2*ht;
        m_crop_box = grow_bbox_to_int(m_crop_box);
      }
      m_crop_box.crop(m_img_bbox);
#endif

      return;
    }

    // We have tabulated point_to_pixel at the mean dem height.
    // Look-up point_to_pixel for the current point by first
    // intersecting the ray from the current point to the camera
    // with the datum at that height. We don't know that ray,
    // so we iterate to find it.
    virtual Vector2 point_to_pixel(Vector3 const& xyz) const{

      if (m_use_semi_approx){
        vw::Mutex::Lock lock(m_camera_mutex);
        g_num_locks++;
        return m_exact_unadjusted_camera->point_to_pixel(xyz);
      }
      
      if (m_use_rpc_approximation) 
        return m_rpc_model->point_to_pixel(xyz);
      
      // TODO: What happens if we use bicubic interpolation?
      InterpolationView<EdgeExtensionView< ImageView< PixelMask<Vector3> >, ConstantEdgeExtension >, BilinearInterpolation> pixel_to_vec_interp
        = interpolate(m_pixel_to_vec_mat, BilinearInterpolation(),
                      ConstantEdgeExtension());

      InterpolationView<EdgeExtensionView< ImageView< PixelMask<Vector2> >, ConstantEdgeExtension >, BilinearInterpolation> point_to_pix_interp
        = interpolate(m_point_to_pix_mat, BilinearInterpolation(),
                      ConstantEdgeExtension());

      Vector3 dir = m_mean_dir;
      Vector2 pix;
      double major_radius = m_geo.datum().semi_major_axis() + m_mean_ht;
      double minor_radius = m_geo.datum().semi_minor_axis() + m_mean_ht;
      for (size_t i = 0; i < 10; i++) {

        Vector3 S = xyz - 1.1*major_radius*dir; // push the point outside the sphere
        if (norm_2(S) <= major_radius) {
          // should not happen. Return the exact solution.
          {
            vw::Mutex::Lock lock(m_camera_mutex);
            g_num_locks++;
            if (g_warning_count < g_max_warning_count) {
              g_warning_count++;
              vw_out(WarningMessage) << "3D point is inside the planet.\n";
            }
            return m_exact_unadjusted_camera->point_to_pixel(xyz);
          }
        }

        Vector3 datum_pt = datum_intersection(major_radius, minor_radius, S, dir);
        Vector3 llh = m_geo.datum().cartesian_to_geodetic(datum_pt);
        Vector2 pt = m_geo.lonlat_to_point(subvector(llh, 0, 2));

        // Indices
        double x = (pt.x() - m_point_box.min().x())/m_approx_table_gridx;
        double y = (pt.y() - m_point_box.min().y())/m_approx_table_gridy;

        bool out_of_range = ( x < 0 || x >= m_pixel_to_vec_mat.cols()-1 ||
                              y < 0 || y >= m_pixel_to_vec_mat.rows()-1 );

        bool out_of_comp_range = (x < m_begX || x >= m_endX-1 ||
                                  y < m_begY || y >= m_endY-1);

        // If we are not out of range, but we need to expand the computed table, do that
        if (!m_stop_growing_range && !out_of_range && out_of_comp_range) {
          vw::Mutex::Lock lock(m_camera_mutex);
          g_num_locks++;
          if (g_warning_count < g_max_warning_count) {
            g_warning_count++;
            vw_out(WarningMessage) << "Pixel outside of computed range. "
                                   << "Growing the computed table." << std::endl;
            vw_out(WarningMessage) << "Start table: " << m_begX << ' ' << m_begY << ' '
                                   << m_endX << ' ' << m_endY << std::endl;
          }
          
          // If we have to expand, do it by a lot
          int extrax = std::max(10, int(0.1*(m_endX - m_begX)));
          int extray = std::max(10, int(0.1*(m_endY - m_begY)));

          int old_begX = m_begX, old_begY = m_begY;
          int old_endX = m_endX, old_endY = m_endY;

          m_begX = std::min(m_begX, int(floor(x))) - extrax; m_begX = std::max(0, m_begX);
          m_begY = std::min(m_begY, int(floor(y))) - extray; m_begY = std::max(0, m_begY);
      
          m_endX = std::max(m_endX, int(ceil(x))) + extrax;
          m_endX = std::min(m_pixel_to_vec_mat.cols()-1, m_endX);

          m_endY = std::max(m_endY, int(ceil(y))) + extray;
          m_endY = std::min(m_pixel_to_vec_mat.rows()-1, m_endY);
      
          if (g_warning_count < g_max_warning_count) {
            vw_out(WarningMessage) << "Updated table: " << m_begX << ' ' << m_begY << ' '
                                   << m_endX << ' ' << m_endY << std::endl;
          }
          comp_entries_in_table();

          // Update this
          out_of_comp_range = (x < m_begX || x >= m_endX-1 ||
                               y < m_begY || y >= m_endY-1);

          // Avoid an infinite loop if we can't grow the table
          if (old_begX == m_begX && old_begY == m_begY &&
              old_endX == m_endX && old_endY == m_endY ) {
            m_stop_growing_range = true;
          }
        }

        if (out_of_range || out_of_comp_range){
          vw::Mutex::Lock lock(m_camera_mutex);
          g_num_locks++;
          if (g_warning_count < g_max_warning_count) {
            g_warning_count++;
            vw_out(WarningMessage) << "Pixel outside of range. Current values and range: "  << ' '
                                   << x << ' ' << y << ' '
                                   << m_pixel_to_vec_mat.cols() << ' ' << m_pixel_to_vec_mat.rows()
                                   << std::endl;
          }
          return m_exact_unadjusted_camera->point_to_pixel(xyz);
        }
        PixelMask<Vector3> masked_dir = pixel_to_vec_interp(x, y);
        PixelMask<Vector2> masked_pix = point_to_pix_interp(x, y);

        if (is_valid(masked_dir) && is_valid(masked_pix)) {
          dir = masked_dir.child();
          pix = masked_pix.child();
        }else{
          {
            vw::Mutex::Lock lock(m_camera_mutex);
            g_num_locks++;
            if (g_warning_count < g_max_warning_count) {
              g_warning_count++;
              vw_out(WarningMessage) << "Invalid ground to camera direction: "
                                     << masked_dir << ' ' << masked_pix << std::endl;
            }
            return m_exact_unadjusted_camera->point_to_pixel(xyz);
          }
        }
      }

      return pix;
    }

    virtual ~ApproxCameraModel(){}
    virtual std::string type() const{ return "ApproxIsis"; }

    virtual Vector3 pixel_to_vector(Vector2 const& pix) const {

      if (m_use_semi_approx) {
        vw::Mutex::Lock lock(m_camera_mutex);
        g_num_locks++;
        return this->exact_unadjusted_camera()->pixel_to_vector(pix);
      }

      if (m_use_rpc_approximation){
        return m_rpc_model->pixel_to_vector(pix);
      }
      
      vw::Mutex::Lock lock(m_camera_mutex);
      g_num_locks++;
      if (g_warning_count < g_max_warning_count) {
        g_warning_count++;
        vw_out(WarningMessage) << "Invoked exact camera model pixel_to_vector for pixel: "
                               << pix << std::endl;
      }
      return this->exact_unadjusted_camera()->pixel_to_vector(pix);
    }

    virtual Vector3 camera_center(Vector2 const& pix) const{
      // It is tricky to approximate the camera center
      //if (m_use_rpc_approximation){
      vw::Mutex::Lock lock(m_camera_mutex);
      g_num_locks++;
      //vw_out(WarningMessage) << "Invoked the camera center function for pixel: "
      //                       << pix << std::endl;
      return this->exact_unadjusted_camera()->camera_center(pix);
      //return m_rpc_model->camera_center(pix);
      //}

#if 0
      // TODO: Is this function invoked? Should just the underlying exact model
      // camera center be used all the time?
      InterpolationView<EdgeExtensionView< ImageView< PixelMask<Vector3> >, ConstantEdgeExtension >, BilinearInterpolation> camera_center_interp
        = interpolate(m_camera_center_mat, BilinearInterpolation(),
                      ConstantEdgeExtension());
      double lx = pix[0] - m_crop_box.min().x();
      double ly = pix[1] - m_crop_box.min().y();
      if (0 <= lx && lx < m_camera_center_mat.cols() - 1 &&
          0 <= ly && ly < m_camera_center_mat.rows() - 1 ) {
        PixelMask<Vector3> ctr = camera_center_interp(lx, ly);
        if (is_valid(ctr))
          return ctr.child();
      }
#endif
      
      {
        // Failed to interpolate
        vw::Mutex::Lock lock(m_camera_mutex);
        g_num_locks++;
        if (g_warning_count < g_max_warning_count) {
          g_warning_count++;
          vw_out(WarningMessage) << "Invoked the camera center function for pixel: "
                                 << pix << std::endl;
        }
        return this->exact_unadjusted_camera()->camera_center(pix);
      }

    }

    virtual Quat camera_pose(Vector2 const& pix) const{
      vw::Mutex::Lock lock(m_camera_mutex);
      g_num_locks++;
      if (g_warning_count < g_max_warning_count) {
        g_warning_count++;
        vw_out(WarningMessage) << "Invoked the camera pose function for pixel: "
                               << pix << std::endl;
      }
      return this->exact_unadjusted_camera()->camera_pose(pix);
    }

  };

  // TODO(oalexan1): Must use the adjusted model in the camera center
  // and camera pose functions!
  // This class provides an approximation for an adjusted ISIS camera
  // model around a current DEM. Unlike the ApproxCameraModel class,
  // here the adjusted camera is approximated, not the unadjusted one,
  // hence the adjustments and the cameras themselves cannot be
  // floated with this class. Keeping the cameras fixed allows the
  // domain of approximation to be narrower so using less memory. The
  // algorithm works by tabulation of point_to_pixel and
  // pixel_to_vector values at the mean dem height.
  class ApproxAdjustedCameraModel: public ApproxBaseCameraModel {
    mutable Vector3 m_mean_dir; // mean vector from camera to ground
    GeoReference m_geo;
    double m_mean_ht;
    mutable ImageView< PixelMask<Vector3> > m_pixel_to_vec_mat;
    mutable ImageView< PixelMask<Vector2> > m_point_to_pix_mat;
    double m_approx_table_gridx, m_approx_table_gridy;
    vw::Mutex& m_camera_mutex;
    Vector2 m_uncompValue;
    mutable int m_begX, m_endX, m_begY, m_endY;
    mutable int m_count;
    
    void comp_entries_in_table() const{
      for (int x = m_begX; x <= m_endX; x++) {
        for (int y = m_begY; y <= m_endY; y++) {
      
          // This will be useful when we invoke this function repeatedly
          if (m_point_to_pix_mat(x, y).child() != m_uncompValue) {
            continue;
          }
      
          Vector2 pt(m_point_box.min().x() + x*m_approx_table_gridx,
                     m_point_box.min().y() + y*m_approx_table_gridy);
          Vector2 lonlat = m_geo.point_to_lonlat(pt);
          Vector3 xyz = m_geo.datum().geodetic_to_cartesian
            (Vector3(lonlat[0], lonlat[1], m_mean_ht));
          bool success = true;
          Vector2 pix;
          Vector3 vec;
          try {
            pix = m_exact_adjusted_camera.point_to_pixel(xyz);
            //if (true || m_img_bbox.contains(pix))  // Need to think more here
            vec = m_exact_adjusted_camera.pixel_to_vector(pix);
            //else
            // success = false;
            
          }catch(...){
            success = false;
          }
          if (success) {
            m_pixel_to_vec_mat(x, y) = vec;
            m_point_to_pix_mat(x, y) = pix;
            m_pixel_to_vec_mat(x, y).validate();
            m_point_to_pix_mat(x, y).validate();
            m_mean_dir += vec; // only when the point projects inside the camera?
            if (m_img_bbox.contains(pix)) 
              m_crop_box.grow(pix);
            m_count++;
          }else{
            m_pixel_to_vec_mat(x, y).invalidate();
            m_point_to_pix_mat(x, y).invalidate();
          }
        }
      }
      
    }
    
  public:

    ApproxAdjustedCameraModel(AdjustedCameraModel const& exact_adjusted_camera,
                              boost::shared_ptr<CameraModel> exact_unadjusted_camera,
                              BBox2i img_bbox, 
                              ImageView<double> const& dem,
                              GeoReference const& geo,
                              double nodata_val,
                              vw::Mutex &camera_mutex):
      ApproxBaseCameraModel(exact_adjusted_camera, exact_unadjusted_camera, img_bbox),
      m_geo(geo), m_camera_mutex(camera_mutex) {

      // Initialize members of the base class
      m_model_is_valid = true;

      int big = 1e+8;
      m_uncompValue = Vector2(-big, -big);
      
      if (dynamic_cast<AdjustedCameraModel*>(exact_unadjusted_camera.get()) != NULL)
        vw_throw( ArgumentErr()
                  << "ApproxAdjustedCameraModel: Expecting an unadjusted camera model.\n");

      // Compute the mean DEM height.
      // We expect all DEM entries to be valid.
      m_mean_ht = 0;
      double num = 0.0;
      for (int col = 0; col < dem.cols(); col++) {
        for (int row = 0; row < dem.rows(); row++) {
          if (dem(col, row) == nodata_val)
            vw_throw( ArgumentErr()
                      << "ApproxAdjustedCameraModel: Expecting a DEM without nodata values.\n");
          m_mean_ht += dem(col, row);
          num += 1.0;
        }
      }
      if (num > 0) m_mean_ht /= num;

      // The area we're supposed to work around
      m_point_box = m_geo.pixel_to_point_bbox(bounding_box(dem));
      double wx = m_point_box.width(), wy = m_point_box.height();
      m_approx_table_gridx = wx/std::max(dem.cols(), 1);
      m_approx_table_gridy = wy/std::max(dem.rows(), 1);

      if (m_approx_table_gridx == 0 || m_approx_table_gridy == 0) {
        vw_throw( ArgumentErr()
                  << "ApproxAdjustedCameraModel: Expecting a positive grid size.\n");
      }

      // Expand the box, as later the DEM will change. 
      double extra = 0.5;
      m_point_box.min().x() -= extra*wx; m_point_box.max().x() += extra*wx;
      m_point_box.min().y() -= extra*wy; m_point_box.max().y() += extra*wy;
      wx = m_point_box.width();
      wy = m_point_box.height();

      vw_out() << "Approximation proj box: " << m_point_box << std::endl;

      // We will tabulate the point_to_pixel function at a multiple of
      // the grid, and we'll use interpolation for anything in
      // between.
      //m_approx_table_gridx /= 2.0; m_approx_table_gridy /= 2.0; // fine
      m_approx_table_gridx *= 2.0; m_approx_table_gridy *= 2.0; // Coarse. Good enough.

      int numx = wx/m_approx_table_gridx;
      int numy = wy/m_approx_table_gridy;

      vw_out() << "Lookup table dimensions: " << numx << ' ' << numy << std::endl;

      m_begX = 0; m_endX = numx-1;
      m_begY = 0; m_endY = numy-1;
      
      //vw_out() << "Size of actually pre-computed table: "
      //           << m_endX - m_begX << ' ' << m_endY - m_begY << std::endl;
      
      // Mark all values as uncomputed and invalid
      m_pixel_to_vec_mat.set_size(numx, numy);
      m_point_to_pix_mat.set_size(numx, numy);
      for (int x = 0; x < numx; x++) {
        for (int y = 0; y < numy; y++) {
          m_point_to_pix_mat(x, y) = m_uncompValue;
          m_point_to_pix_mat(x, y).invalidate();
        }
      }
      
      // Fill in the table. Find along the way the mean direction from
      // the camera to the ground. Invalid values will be masked.
      m_count = 0;
      m_mean_dir = Vector3();
      comp_entries_in_table();
      m_mean_dir /= std::max(1, m_count);
      m_mean_dir = m_mean_dir/norm_2(m_mean_dir);
      
      m_crop_box.crop(m_img_bbox);

      return;
    }

    // We have tabulated point_to_pixel at the mean dem height.
    // Look-up point_to_pixel for the current point by first
    // intersecting the ray from the current point to the camera
    // with the datum at that height. We don't know that ray,
    // so we iterate to find it.
    virtual Vector2 point_to_pixel(Vector3 const& xyz) const{

      // TODO: What happens if we use bicubic interpolation?
      InterpolationView<EdgeExtensionView< ImageView< PixelMask<Vector3> >, ConstantEdgeExtension >, BilinearInterpolation> pixel_to_vec_interp
        = interpolate(m_pixel_to_vec_mat, BilinearInterpolation(),
                      ConstantEdgeExtension());

      InterpolationView<EdgeExtensionView< ImageView< PixelMask<Vector2> >, ConstantEdgeExtension >, BilinearInterpolation> point_to_pix_interp
        = interpolate(m_point_to_pix_mat, BilinearInterpolation(),
                      ConstantEdgeExtension());

      Vector3 dir = m_mean_dir;
      Vector2 pix;
      double major_radius = m_geo.datum().semi_major_axis() + m_mean_ht;
      double minor_radius = m_geo.datum().semi_minor_axis() + m_mean_ht;
      for (size_t i = 0; i < 10; i++) {

        Vector3 S = xyz - 1.1*major_radius*dir; // push the point outside the sphere
        if (norm_2(S) <= major_radius) {
          // should not happen. Return the exact solution.
          {
            vw::Mutex::Lock lock(m_camera_mutex);
            g_num_locks++;
            if (g_warning_count < g_max_warning_count) {
              g_warning_count++;
              vw_out(WarningMessage) << "3D point is inside the planet.\n";
            }
            return m_exact_adjusted_camera.point_to_pixel(xyz);
          }
        }

        Vector3 datum_pt = datum_intersection(major_radius, minor_radius, S, dir);
        Vector3 llh = m_geo.datum().cartesian_to_geodetic(datum_pt);
        Vector2 pt = m_geo.lonlat_to_point(subvector(llh, 0, 2));

        // Indices
        double x = (pt.x() - m_point_box.min().x())/m_approx_table_gridx;
        double y = (pt.y() - m_point_box.min().y())/m_approx_table_gridy;

        bool out_of_range = (x < m_begX || x >= m_endX-1 ||
                             y < m_begY || y >= m_endY-1);

        // If out of range, return the exact result. This should be very slow.
        // The hope is that it will be very rare.
        if (out_of_range){
          vw::Mutex::Lock lock(m_camera_mutex);
          g_num_locks++;
          if (g_warning_count < g_max_warning_count) {
            g_warning_count++;
            vw_out(WarningMessage) << "Pixel outside of range. Current values and range: "  << ' '
                                   << x << ' ' << y << ' '
                                   << m_pixel_to_vec_mat.cols() << ' ' << m_pixel_to_vec_mat.rows()
                                   << std::endl;
          }
          return m_exact_adjusted_camera.point_to_pixel(xyz);
        }
        
        PixelMask<Vector3> masked_dir = pixel_to_vec_interp(x, y);
        PixelMask<Vector2> masked_pix = point_to_pix_interp(x, y);
        if (is_valid(masked_dir) && is_valid(masked_pix)) {
          dir = masked_dir.child();
          pix = masked_pix.child();
        }else{
          {
            vw::Mutex::Lock lock(m_camera_mutex);
            g_num_locks++;
            if (g_warning_count < g_max_warning_count) {
              g_warning_count++;
              vw_out(WarningMessage) << "Invalid ground to camera direction: "
                                     << masked_dir << ' ' << masked_pix << std::endl;
            }
            return m_exact_adjusted_camera.point_to_pixel(xyz);
          }
        }
      }

      return pix;
    }

    virtual ~ApproxAdjustedCameraModel(){}
    virtual std::string type() const{ return "ApproxAdjustedIsis"; }

    virtual Vector3 pixel_to_vector(Vector2 const& pix) const {

      vw::Mutex::Lock lock(m_camera_mutex);
      g_num_locks++;
      if (g_warning_count < g_max_warning_count) {
        g_warning_count++;
        vw_out(WarningMessage) << "Invoked exact camera model pixel_to_vector for pixel: "
                               << pix << std::endl;
      }
      // TODO(oalexan1): Put here the exact adjusted camera!
      return this->exact_unadjusted_camera()->pixel_to_vector(pix);
    }

    virtual Vector3 camera_center(Vector2 const& pix) const{
      // It is tricky to approximate the camera center
      // TODO(oalexan1): Must apply the adjustment here?
      vw::Mutex::Lock lock(m_camera_mutex);
      g_num_locks++;
      // TODO(oalexan1): Put here the exact adjusted camera!
      return this->exact_unadjusted_camera()->camera_center(pix);
    }

    virtual Quat camera_pose(Vector2 const& pix) const{
      // TODO(oalexan1): Must apply the adjustment here?!!!
      vw::Mutex::Lock lock(m_camera_mutex);
      g_num_locks++;
      if (g_warning_count < g_max_warning_count) {
        g_warning_count++;
        vw_out(WarningMessage) << "Invoked the camera pose function for pixel: "
                               << pix << std::endl;
      }
      // TODO(oalexan1): Put here the exact adjusted camera!
      return this->exact_unadjusted_camera()->camera_pose(pix);
    }

  };
  
}}

// Get the memory usage for the given process. This is for debugging, not used
// in production code. It does not work on OSX.
void callTop() {

  std::ostringstream os;
  int pid = getpid();
  os << pid;
  
  std::string cmd = "top -b -n 1 | grep -i ' sfs' | grep -i '" + os.str() + "'";
  std::string ans = vw::exec_cmd(cmd.c_str());
  vw_out() << "Memory usage: " << cmd << " " << ans << std::endl;
}


// Compute mean and standard deviation of two images. Do it where both are valid.
template <class ImageT>
void compute_image_stats(ImageT const& I1, ImageT const& I2,
                         double & mean1, double & stdev1,
                         double & mean2, double & stdev2) {

  if (I1.cols() != I2.cols() || I1.rows() != I2.rows()) 
    vw_throw(ArgumentErr() << "Expecting two input images of same size.\n");
  
  mean1 = 0; stdev1 = 0;
  mean2 = 0; stdev2 = 0;
  
  double sum1 = 0.0, sum2 = 0.0, sum1_sq = 0.0, sum2_sq = 0.0, count = 0.0;
  for (int col = 0; col < I1.cols(); col++){
    for (int row = 0; row < I1.rows(); row++){
      
      if (!is_valid(I1(col, row)) || !is_valid(I2(col, row))) continue;
                    
      count++;
      
      double val1 = I1(col, row); sum1 += val1; sum1_sq += val1*val1;
      double val2 = I2(col, row); sum2 += val2; sum2_sq += val2*val2;
    }
  }

  if (count > 0){
    mean1 = sum1/count; stdev1 = sqrt(sum1_sq/count - mean1*mean1);
    mean2 = sum2/count; stdev2 = sqrt(sum2_sq/count - mean2*mean2);
  }

}

struct Options : public vw::GdalWriteOptions {
  std::string input_dems_str, image_list, camera_list, out_prefix, stereo_session, bundle_adjust_prefix;
  std::vector<std::string> input_dems, input_images, input_cameras;
  std::string shadow_thresholds, custom_shadow_threshold_list, max_valid_image_vals, skip_images_str, image_exposure_prefix, model_coeffs_prefix, model_coeffs, image_haze_prefix, sun_positions_list;
  std::vector<float> shadow_threshold_vec, max_valid_image_vals_vec;
  std::vector<double> image_exposures_vec;
  std::vector<std::vector<double>> image_haze_vec;
  std::vector<double> model_coeffs_vec;
  std::vector<std::set<int>> skip_images;
  int max_iterations, max_coarse_iterations, reflectance_type, coarse_levels,
    blending_dist, min_blend_size, num_haze_coeffs;
  bool float_albedo, float_exposure, float_cameras, float_all_cameras, model_shadows,
    save_computed_intensity_only, estimate_slope_errors, estimate_height_errors,
    compute_exposures_only,
    save_dem_with_nodata, use_approx_camera_models, use_approx_adjusted_camera_models,
    use_rpc_approximation, use_semi_approx,
    crop_input_images, allow_borderline_data, float_dem_at_boundary, boundary_fix,
    fix_dem, float_reflectance_model, float_sun_position, query, save_sparingly,
    float_haze;
    
  double smoothness_weight, steepness_factor, curvature_in_shadow,
    curvature_in_shadow_weight,
    lit_curvature_dist, shadow_curvature_dist, gradient_weight,
    blending_power, integrability_weight, smoothness_weight_pq, init_dem_height,
    nodata_val, initial_dem_constraint_weight, albedo_constraint_weight,
    albedo_robust_threshold,
    camera_position_step_size, rpc_penalty_weight, rpc_max_error,
    unreliable_intensity_threshold, robust_threshold, shadow_threshold;
  vw::BBox2 crop_win;
  vw::Vector2 height_error_params;
  
  Options():max_iterations(0), max_coarse_iterations(0), reflectance_type(0),
            coarse_levels(0), blending_dist(0), blending_power(2.0),
            min_blend_size(0), num_haze_coeffs(0),
            float_albedo(false), float_exposure(false), float_cameras(false),
            float_all_cameras(false),
            model_shadows(false), 
            save_computed_intensity_only(false),
            estimate_slope_errors(false),
            estimate_height_errors(false),
            compute_exposures_only(false),
            save_dem_with_nodata(false),
            use_approx_camera_models(false),
            use_approx_adjusted_camera_models(false),
            use_rpc_approximation(false),
            use_semi_approx(false),
            crop_input_images(false),
            allow_borderline_data(false), 
            float_dem_at_boundary(false), boundary_fix(false), fix_dem(false),
            float_reflectance_model(false), float_sun_position(false),
            query(false), save_sparingly(false), float_haze(false),
            smoothness_weight(0), steepness_factor(1.0),
            curvature_in_shadow(0), curvature_in_shadow_weight(0.0),
            lit_curvature_dist(0.0), shadow_curvature_dist(0.0),
            gradient_weight(0.0), integrability_weight(0), smoothness_weight_pq(0),
            initial_dem_constraint_weight(0.0),
            albedo_constraint_weight(0.0), albedo_robust_threshold(0.0),
            camera_position_step_size(1.0), rpc_penalty_weight(0.0),
            rpc_max_error(0.0),
            unreliable_intensity_threshold(0.0),
            crop_win(BBox2i(0, 0, 0, 0)){}
};

struct GlobalParams{
  int reflectanceType;
  // Two parameters used in the formula for the Lunar-Lambertian
  // reflectance
  double phaseCoeffC1, phaseCoeffC2;
};

struct ModelParams {
  vw::Vector3 sunPosition; //relative to the center of the Moon
  ModelParams(){}
  ~ModelParams(){}
};

// Make the reflectance nonlinear using a rational function
double nonlin_reflectance(double reflectance, double exposure,
                          double steepness_factor,
                          double const* haze, int num_haze_coeffs){

  // Make the exposure smaller. This will result in higher reflectance
  // to compensate, as intensity = exposure * reflectance, hence
  // steeper terrain. Things become more complicated if the haze
  // and nonlinear reflectance is modeled.
  exposure /= steepness_factor;
  
  double r = reflectance; // for short
  if (num_haze_coeffs == 0) return (exposure*r);
  if (num_haze_coeffs == 1) return (exposure*r + haze[0]);
  if (num_haze_coeffs == 2) return (exposure*r + haze[0])/(haze[1]*r + 1);
  if (num_haze_coeffs == 3) return (haze[2]*r*r + exposure*r + haze[0])/(haze[1]*r + 1);
  if (num_haze_coeffs == 4) return (haze[2]*r*r + exposure*r + haze[0])/(haze[3]*r*r + haze[1]*r + 1);
  if (num_haze_coeffs == 5) return (haze[4]*r*r*r + haze[2]*r*r + exposure*r + haze[0])/(haze[3]*r*r + haze[1]*r + 1);
  if (num_haze_coeffs == 6) return (haze[4]*r*r*r + haze[2]*r*r + exposure*r + haze[0])/(haze[5]*r*r*r + haze[3]*r*r + haze[1]*r + 1);
    
  vw_throw(ArgumentErr() << "Invalid value for the number of haze coefficients.\n");
  return 0;
}
                          
enum {NO_REFL = 0, LAMBERT, LUNAR_LAMBERT, HAPKE, ARBITRARY_MODEL, CHARON};

// computes the Lambertian reflectance model (cosine of the light
// direction and the normal to the Moon) Vector3 sunpos: the 3D
// coordinates of the Sun relative to the center of the Moon Vector2
// lon_lat is a 2D vector. First element is the longitude and the
// second the latitude.
//author Ara Nefian
double
computeLambertianReflectanceFromNormal(Vector3 sunPos, Vector3 xyz,
                                       Vector3 normal) {
  double reflectance;
  Vector3 sunDirection = normalize(sunPos-xyz);

  reflectance = sunDirection[0]*normal[0] + sunDirection[1]*normal[1] + sunDirection[2]*normal[2];

  return reflectance;
}


double computeLunarLambertianReflectanceFromNormal(Vector3 const& sunPos,
                                                   Vector3 const& viewPos,
                                                   Vector3 const& xyz,
                                                   Vector3 const& normal,
                                                   double phaseCoeffC1,
                                                   double phaseCoeffC2,
                                                   double & alpha,
                                                   const double * reflectance_model_coeffs) {
  double reflectance;
  double L;

  double len = dot_prod(normal, normal);
  if (abs(len - 1.0) > 1.0e-4){
    std::cerr << "Error: Expecting unit normal in the reflectance computation, in "
              << __FILE__ << " at line " << __LINE__ << std::endl;
    exit(1);
  }

  //compute /mu_0 = cosine of the angle between the light direction and the surface normal.
  //sun coordinates relative to the xyz point on the Moon surface
  Vector3 sunDirection = normalize(sunPos-xyz);
  double mu_0 = dot_prod(sunDirection, normal);

  //double tol = 0.3;
  //if (mu_0 < tol){
  //  // Sun is too low, reflectance is too close to 0, the albedo will be inaccurate
  //  return 0.0;
  // }

  //compute  /mu = cosine of the angle between the viewer direction and the surface normal.
  //viewer coordinates relative to the xyz point on the Moon surface
  Vector3 viewDirection = normalize(viewPos-xyz);
  double mu = dot_prod(viewDirection,normal);

  //compute the phase angle (alpha) between the viewing direction and the light source direction
  double deg_alpha;
  double cos_alpha;

  cos_alpha = dot_prod(sunDirection, viewDirection);
  if ((cos_alpha > 1)||(cos_alpha< -1)){
    printf("cos_alpha error\n");
  }

  alpha     = acos(cos_alpha);  // phase angle in radians
  deg_alpha = alpha*180.0/M_PI; // phase angle in degrees

  //printf("deg_alpha = %f\n", deg_alpha);

  //Bob Gaskell's model
  //L = exp(-deg_alpha/60.0);

  //Alfred McEwen's model
  double O = reflectance_model_coeffs[0]; // 1
  double A = reflectance_model_coeffs[1]; //-0.019;
  double B = reflectance_model_coeffs[2]; // 0.000242;//0.242*1e-3;
  double C = reflectance_model_coeffs[3]; // -0.00000146;//-1.46*1e-6;

  L = O + A*deg_alpha + B*deg_alpha*deg_alpha + C*deg_alpha*deg_alpha*deg_alpha;
 
  //printf(" deg_alpha = %f, L = %f\n", deg_alpha, L);

  //if (mu_0 < 0.0){
  //  return 0.0;
  // }

  //  if (mu < 0.0){ //emission angle is > 90
  //  mu = 0.0;
  //}

  //if (mu_0 + mu == 0){
  //  //printf("negative reflectance\n");
  //  return 0.0;
  //}
  //else{
  reflectance = 2*L*mu_0/(mu_0+mu) + (1-L)*mu_0;
  //}
  
  //if (mu < 0 || mu_0 < 0 || mu_0 + mu <= 0 ||  reflectance <= 0 || reflectance != reflectance){
  if (mu_0 + mu == 0 || reflectance != reflectance){
    return 0.0;
  }

  // Attempt to compensate for points on the terrain being too bright
  // if the sun is behind the spacecraft as seen from those points.

  //reflectance *= std::max(0.4, exp(-alpha*alpha));
  reflectance *= ( exp(-phaseCoeffC1*alpha) + phaseCoeffC2 );

  return reflectance;
}

// Hapke's model.
// See: An Experimental Study of Light Scattering by Large, Irregular Particles
// Audrey F. McGuire, Bruce W. Hapke. 1995. The reflectance used is R(g), in equation
// above Equation 21. The p(g) function is given by Equation (14), yet this one uses
// an old convention. The updated p(g) is given in:
// Spectrophotometric properties of materials observed by Pancam on the Mars Exploration Rovers: 1.
// Spirit. JR Johnson, 2006.
// We Use the two-term p(g), and the parameter c, not c'=1-c.
// We also use the values of w(=omega), b, and c from that table.
// Note that we use the updated Hapke model, having the term B(g). This one is given in
// "Modeling spectral and bidirectional reflectance", Jacquemoud, 1992. It has the params
// B0 and h.
// The ultimate reference is probably Hapke, 1986, having all pieces in one place, but
// that one is not available. 
// We use mostly the parameter values for omega, b, c, B0 and h from:
// Surface reflectance of Mars observed by CRISM/MRO: 2.
// Estimation of surface photometric properties in Gusev Crater and Meridiani Planum by J. Fernando. 
// See equations (1), (2) and (4) in that paper.
// Example values for the params: w=omega=0.68, b=0.17, c=0.62, B0=0.52, h=0.52.
// But we don't use equation (3) from that paper, we use instead what they call the formula H93,
// which is the H(x) from McGuire and Hapke 1995 mentioned above.
// See the complete formulas below.
double computeHapkeReflectanceFromNormal(Vector3 const& sunPos,
                                         Vector3 const& viewPos,
                                         Vector3 const& xyz,
                                         Vector3 const& normal,
                                         double phaseCoeffC1,
                                         double phaseCoeffC2,
                                         double & alpha,
                                         const double * reflectance_model_coeffs) {

  double len = dot_prod(normal, normal);
  if (abs(len - 1.0) > 1.0e-4){
    std::cerr << "Error: Expecting unit normal in the reflectance computation, in "
              << __FILE__ << " at line " << __LINE__ << std::endl;
    exit(1);
  }

  //compute mu_0 = cosine of the angle between the light direction and the surface normal.
  //sun coordinates relative to the xyz point on the Moon surface
  Vector3 sunDirection = normalize(sunPos-xyz);
  double mu_0 = dot_prod(sunDirection, normal);

  //compute mu = cosine of the angle between the viewer direction and the surface normal.
  //viewer coordinates relative to the xyz point on the Moon surface
  Vector3 viewDirection = normalize(viewPos-xyz);
  double mu = dot_prod(viewDirection,normal);

  //compute the phase angle (g) between the viewing direction and the light source direction
  // in radians
  double cos_g = dot_prod(sunDirection, viewDirection);
  double g = acos(cos_g);  // phase angle in radians

  // Hapke params
  double omega = std::abs(reflectance_model_coeffs[0]); // also known as w
  double b     = std::abs(reflectance_model_coeffs[1]);
  double c     = std::abs(reflectance_model_coeffs[2]);
  // The older Hapke model lacks the B0 and h terms
  double B0    = std::abs(reflectance_model_coeffs[3]);
  double h     = std::abs(reflectance_model_coeffs[4]);   

  double J = 1.0; // does not matter, we'll factor out the constant scale as camera exposures anyway
  
  // The P(g) term
  double Pg 
    = (1.0 - c) * (1.0 - b*b) / pow(1.0 + 2.0*b*cos_g + b*b, 1.5)
    + c         * (1.0 - b*b) / pow(1.0 - 2.0*b*cos_g + b*b, 1.5);
    
  // The B(g) term
  double Bg = B0 / ( 1.0 + (1.0/h)*tan(g/2.0) );

  double H_mu0 = (1.0 + 2*mu_0) / (1.0 + 2*mu_0 * sqrt(1.0 - omega));
  double H_mu  = (1.0 + 2*mu  ) / (1.0 + 2*mu   * sqrt(1.0 - omega));

  // The reflectance
  double R = (J*omega/4.0/M_PI) * ( mu_0/(mu_0+mu) ) * ( (1.0 + Bg)*Pg + H_mu0*H_mu - 1.0 );
  
  return R;
}

// Use the following model:
// Reflectance = f(alpha) * A * mu_0 /(mu_0 + mu) + (1-A) * mu_0
// The value of A is either 1 (the so-called lunar-model), or A=0.7.
// f(alpha) = 0.63.
double computeCharonReflectanceFromNormal(Vector3 const& sunPos,
                                          Vector3 const& viewPos,
                                          Vector3 const& xyz,
                                          Vector3 const& normal,
                                          double phaseCoeffC1,
                                          double phaseCoeffC2,
                                          double & alpha,
                                          const double * reflectance_model_coeffs) {

  double len = dot_prod(normal, normal);
  if (abs(len - 1.0) > 1.0e-4){
    std::cerr << "Error: Expecting unit normal in the reflectance computation, in "
              << __FILE__ << " at line " << __LINE__ << std::endl;
    exit(1);
  }

  //compute mu_0 = cosine of the angle between the light direction and the surface normal.
  //sun coordinates relative to the xyz point on the Moon surface
  Vector3 sunDirection = normalize(sunPos-xyz);
  double mu_0 = dot_prod(sunDirection, normal);

  //compute mu = cosine of the angle between the viewer direction and the surface normal.
  //viewer coordinates relative to the xyz point on the Moon surface
  Vector3 viewDirection = normalize(viewPos-xyz);
  double mu = dot_prod(viewDirection,normal);

  // Charon model params
  double A       = std::abs(reflectance_model_coeffs[0]); // albedo 
  double f_alpha = std::abs(reflectance_model_coeffs[1]); // phase function 

  double reflectance = f_alpha*A*mu_0 / (mu_0 + mu) + (1.0 - A)*mu_0;
  
  if (mu_0 + mu == 0 || reflectance != reflectance){
    return 0.0;
  }

  return reflectance;
}

double computeArbitraryLambertianReflectanceFromNormal(Vector3 const& sunPos,
                                                       Vector3 const& viewPos,
                                                       Vector3 const& xyz,
                                                       Vector3 const& normal,
                                                       double phaseCoeffC1,
                                                       double phaseCoeffC2,
                                                       double & alpha,
                                                       const double * reflectance_model_coeffs) {
  double reflectance;

  double len = dot_prod(normal, normal);
  if (abs(len - 1.0) > 1.0e-4){
    std::cerr << "Error: Expecting unit normal in the reflectance computation, in "
              << __FILE__ << " at line " << __LINE__ << std::endl;
    exit(1);
  }

  //compute /mu_0 = cosine of the angle between the light direction and the surface normal.
  //sun coordinates relative to the xyz point on the Moon surface
  //Vector3 sunDirection = -normalize(sunPos-xyz);
  Vector3 sunDirection = normalize(sunPos-xyz);
  double mu_0 = dot_prod(sunDirection, normal);

  //double tol = 0.3;
  //if (mu_0 < tol){
  //  // Sun is too low, reflectance is too close to 0, the albedo will be inaccurate
  //  return 0.0;
  // }

  //compute  /mu = cosine of the angle between the viewer direction and the surface normal.
  //viewer coordinates relative to the xyz point on the Moon surface
  Vector3 viewDirection = normalize(viewPos-xyz);
  double mu = dot_prod(viewDirection,normal);

  //compute the phase angle (alpha) between the viewing direction and the light source direction
  double deg_alpha;
  double cos_alpha;

  cos_alpha = dot_prod(sunDirection,viewDirection);
  if ((cos_alpha > 1)||(cos_alpha< -1)){
    printf("cos_alpha error\n");
  }

  alpha     = acos(cos_alpha);  // phase angle in radians
  deg_alpha = alpha*180.0/M_PI; // phase angle in degrees

  //printf("deg_alpha = %f\n", deg_alpha);

  //Bob Gaskell's model
  //L = exp(-deg_alpha/60.0);

  //Alfred McEwen's model
  double O1 = reflectance_model_coeffs[0]; // 1
  double A1 = reflectance_model_coeffs[1]; // -0.019;
  double B1 = reflectance_model_coeffs[2]; // 0.000242;//0.242*1e-3;
  double C1 = reflectance_model_coeffs[3]; // -0.00000146;//-1.46*1e-6;
  double D1 = reflectance_model_coeffs[4]; 
  double E1 = reflectance_model_coeffs[5]; 
  double F1 = reflectance_model_coeffs[6]; 
  double G1 = reflectance_model_coeffs[7]; 

  double O2 = reflectance_model_coeffs[8];  // 1
  double A2 = reflectance_model_coeffs[9];  // -0.019;
  double B2 = reflectance_model_coeffs[10]; // 0.000242;//0.242*1e-3;
  double C2 = reflectance_model_coeffs[11]; // -0.00000146;//-1.46*1e-6;
  double D2 = reflectance_model_coeffs[12]; 
  double E2 = reflectance_model_coeffs[13]; 
  double F2 = reflectance_model_coeffs[14]; 
  double G2 = reflectance_model_coeffs[15]; 
  
  double L1 = O1 + A1*deg_alpha + B1*deg_alpha*deg_alpha + C1*deg_alpha*deg_alpha*deg_alpha;
  double K1 = D1 + E1*deg_alpha + F1*deg_alpha*deg_alpha + G1*deg_alpha*deg_alpha*deg_alpha;
  if (K1 == 0) K1 = 1;
    
  double L2 = O2 + A2*deg_alpha + B2*deg_alpha*deg_alpha + C2*deg_alpha*deg_alpha*deg_alpha;
  double K2 = D2 + E2*deg_alpha + F2*deg_alpha*deg_alpha + G2*deg_alpha*deg_alpha*deg_alpha;
  if (K2 == 0) K2 = 1;
  
  //printf(" deg_alpha = %f, L = %f\n", deg_alpha, L);

  //if (mu_0 < 0.0){
  //  return 0.0;
  // }

  //  if (mu < 0.0){ //emission angle is > 90
  //  mu = 0.0;
  //}

  //if (mu_0 + mu == 0){
  //  //printf("negative reflectance\n");
  //  return 0.0;
  //}
  //else{
  reflectance = 2*L1*mu_0/(mu_0+mu)/K1 + (1-L2)*mu_0/K2;
  //}
  
  //if (mu < 0 || mu_0 < 0 || mu_0 + mu <= 0 ||  reflectance <= 0 || reflectance != reflectance){
  if (mu_0 + mu == 0 || reflectance != reflectance){
    return 0.0;
  }

  // Attempt to compensate for points on the terrain being too bright
  // if the sun is behind the spacecraft as seen from those points.

  //reflectance *= std::max(0.4, exp(-alpha*alpha));
  reflectance *= ( exp(-phaseCoeffC1*alpha) + phaseCoeffC2 );

  return reflectance;
}

double ComputeReflectance(Vector3 const& cameraPosition,
                          Vector3 const& normal, Vector3 const& xyz,
                          ModelParams const& input_img_params,
                          GlobalParams const& global_params,
                          double & phase_angle,
                          const double * reflectance_model_coeffs) {
  double input_img_reflectance;

  switch ( global_params.reflectanceType )
    {
    case LUNAR_LAMBERT:
      input_img_reflectance
        = computeLunarLambertianReflectanceFromNormal(input_img_params.sunPosition,
                                                      cameraPosition,
                                                      xyz,  normal,
                                                      global_params.phaseCoeffC1,
                                                      global_params.phaseCoeffC2,
                                                      phase_angle, // output
                                                      reflectance_model_coeffs);
      break;
    case ARBITRARY_MODEL:
      input_img_reflectance
        = computeArbitraryLambertianReflectanceFromNormal(input_img_params.sunPosition,
                                                          cameraPosition,
                                                          xyz,  normal,
                                                          global_params.phaseCoeffC1,
                                                          global_params.phaseCoeffC2,
                                                          phase_angle, // output
                                                          reflectance_model_coeffs);
      break;
    case HAPKE:
      input_img_reflectance
        = computeHapkeReflectanceFromNormal(input_img_params.sunPosition,
                                            cameraPosition,
                                            xyz,  normal,
                                            global_params.phaseCoeffC1,
                                            global_params.phaseCoeffC2,
                                            phase_angle, // output
                                            reflectance_model_coeffs);
      break;
    case CHARON:
      input_img_reflectance
        = computeCharonReflectanceFromNormal(input_img_params.sunPosition,
                                             cameraPosition,
                                             xyz,  normal,
                                             global_params.phaseCoeffC1,
                                             global_params.phaseCoeffC2,
                                             phase_angle, // output
                                             reflectance_model_coeffs);
      break;
    case LAMBERT:
      input_img_reflectance
        = computeLambertianReflectanceFromNormal(input_img_params.sunPosition,
                                                 xyz, normal);
      break;

    default:
      input_img_reflectance = 1;
    }

  return input_img_reflectance;
}

// Use this struct to keep track of height errors.
struct HeightErrEstim {

  HeightErrEstim(int num_cols, int num_rows, int num_height_samples_in,
                 double max_height_error_in, double nodata_height_val_in,
                 ImageView<double> * albedo_in,
                 Options * opt_in) {
    num_height_samples = num_height_samples_in; // TODO(oalexan1): This must be a parameter
    max_height_error   = max_height_error_in;   // TODO(oalexan1): This must be a parameter
    nodata_height_val  = nodata_height_val_in;
    
    albedo = albedo_in;
    opt = opt_in;
    
    image_iter = 0; // will be modified later

    height_error_vec.set_size(num_cols, num_rows);
    for (int col = 0; col < num_cols; col++) {
      for (int row = 0; row < num_rows; row++) {
        height_error_vec(col, row)[0] = -max_height_error;
        height_error_vec(col, row)[1] =  max_height_error;
      }
    }
  }
  
  int num_height_samples;
  ImageView<double> * albedo;
  Options * opt;
  ImageView<Vector2> height_error_vec;
  int image_iter;
  double max_height_error;
  double nodata_height_val;
};

// Use this struct to keep track of slope errors.
struct SlopeErrEstim {

  SlopeErrEstim(int num_cols, int num_rows, int num_a_samples_in, int num_b_samples_in,
                ImageView<double> * albedo_in, Options * opt_in) {
    num_a_samples = num_a_samples_in;
    num_b_samples = num_b_samples_in;
    albedo = albedo_in;
    opt = opt_in;

    image_iter = 0; // will be modified later

    // The maximum possible deviation from the normal in degrees
    max_angle = 90.0; 
    
    slope_errs.resize(num_cols);
    for (int col = 0; col < num_cols; col++) {
      slope_errs[col].resize(num_rows);
      for (int row = 0; row < num_rows; row++) {
        slope_errs[col][row].resize(num_b_samples, max_angle);
      }
    }
  }
  
  int num_a_samples, num_b_samples;
  ImageView<double> * albedo;
  Options * opt;
  std::vector< std::vector< std::vector<double> > > slope_errs;
  int image_iter;
  double max_angle;
};

// Given the normal (slope) to the SfS DEM, find how different
// a slope can be from this before the computed intensity
// due to that slope is bigger than max_intensity_err.
void estimateSlopeError(Vector3 const& cameraPosition,
                        Vector3 const& normal, Vector3 const& xyz,
                        ModelParams const& local_model_params,
                        GlobalParams const& global_params,
                        const double * reflectance_model_coeffs,
                        double meas_intensity,
                        double max_intensity_err,
                        int col, int row, int image_iter,
                        Options & opt,
                        ImageView<double> & albedo,
                        SlopeErrEstim * slopeErrEstim){
  
  // Find the angle u from the normal to the z axis, and the angle v
  // from the x axis to the projection of the normal in the xy plane.
  double u = acos(normal[2]);
  double v = 0.0;
  if (normal[0] != 0.0 || normal[1] != 0.0) 
    v = atan2(normal[1], normal[0]);

  double cv = cos(v), sv = sin(v), cu = cos(u), su = sin(u);
  Vector3 n(cv*su, sv*su, cu);

  // Sanity check, these angles should give us back the normal
  if (norm_2(normal - n) > 1e-8) 
    vw_throw( LogicErr() << "Book-keeping error in slope estimation.\n" );
    
  // Find the rotation R that transforms the vector (0, 0, 1) to the normal
  vw::Matrix3x3 R1, R2, R;
  
  R1[0][0] = cv;  R1[0][1] = -sv; R1[0][2] = 0;
  R1[1][0] = sv;  R1[1][1] =  cv; R1[1][2] = 0;
  R1[2][0] = 0;   R1[2][1] =  0;  R1[2][2] = 1;
  
  R2[0][0] = cu;  R2[0][1] =  0;  R2[0][2] = su;
  R2[1][0] = 0;   R2[1][1] =  1;  R2[1][2] = 0;
  R2[2][0] = -su; R2[2][1] =  0;  R2[2][2] = cu;

  R = R1 * R2;

  // We must have R * n0 = n
  Vector3 n0(0, 0, 1);
  if (norm_2(R*n0 - n) > 1e-8) 
    vw_throw( LogicErr() << "Book-keeping error in slope estimation.\n" );
  
  int num_a_samples = slopeErrEstim->num_a_samples;
  int num_b_samples = slopeErrEstim->num_b_samples;

  int num_cols = slopeErrEstim->slope_errs.size();
  int num_rows = slopeErrEstim->slope_errs[0].size();
  int num_b_samples2 = slopeErrEstim->slope_errs[0][0].size();

  if (num_b_samples != num_b_samples2)
    vw_throw( LogicErr()
              << "Book-keeping failure in estimating the slope error!\n");
  
  // Sample the set of unit vectors w which make the angle 'a' with
  // the normal. For that, start with w having angle 'a' with the z
  // axis, and angle 'b' between the projection of w onto the xy plane
  // and the x axis. Then apply the rotation R to it which will make
  // the angle between w and the normal be 'a'. By varying 'b' we will
  // sample all such angles.
  double deg2rad = M_PI/180.0;
  for (int b_iter = 0; b_iter < num_b_samples; b_iter++) {
    
    double b = 360.0 * double(b_iter)/num_b_samples;
    double cb = cos(deg2rad * b), sb = sin(deg2rad * b);
    
    for (int a_iter = 0; a_iter < num_a_samples; a_iter++) {
      
      double a = 90.0 * double(a_iter)/num_a_samples;

      if (slopeErrEstim->slope_errs[col][row][b_iter] < a) {
        // We already determined that the slope error can't be as big as
        // a, so there is no point to explore bigger angles
        break;
      }

      double ca = cos(deg2rad * a), sa = sin(deg2rad * a);

      Vector3 w(cb*sa, sb*sa, ca);
      w = R*w;

      // Compute here dot product from w to n. Should be cos(a) for all b.
      double prod = dot_prod(w, normal);
      if (std::abs(prod - ca) > 1e-8)
        vw_throw( LogicErr() << "Book-keeping error in slope estimation.\n" );

      // Compute the reflectance with the given normal
      double phase_angle = 0.0;
      PixelMask<double> reflectance = ComputeReflectance(cameraPosition,
                                                         w, xyz, local_model_params,
                                                         global_params, phase_angle,
                                                         reflectance_model_coeffs);
      reflectance.validate();

      double comp_intensity = albedo(col, row) *
        nonlin_reflectance(reflectance, opt.image_exposures_vec[image_iter],
                           opt.steepness_factor,
                           &opt.image_haze_vec[image_iter][0], opt.num_haze_coeffs);

      if (std::abs(comp_intensity - meas_intensity) > max_intensity_err) {
        // We exceeded the error budget, hence this is an upper bound on the slope
        slopeErrEstim->slope_errs[col][row][b_iter] = a;
        break;
      }
      
    }
  }
}

// Given the normal (height) to the SfS DEM, find how different
// a height can be from this before the computed intensity
// due to that height is bigger than max_intensity_err.
void estimateHeightError(ImageView<double> const& dem,
                         cartography::GeoReference const& geo,
                         Vector3 const& cameraPosition,
                         ModelParams const& local_model_params,
                         GlobalParams const& global_params,
                         const double * reflectance_model_coeffs,
                         double meas_intensity,
                         double max_intensity_err,
                         int col, int row, int image_iter,
                         Options & opt,
                         ImageView<double> & albedo,
                         HeightErrEstim * heightErrEstim){

  // Look at the neighbors
  int cols[] = {col - 1, col,     col,     col + 1};
  int rows[] = {row,     row - 1, row + 1, row};
  
  for (int it = 0; it < 4; it++) {

    int colx = cols[it], rowx = rows[it];

    // Can't be at edges as need to compute normals
    if (colx <= 0 || rowx <= 0 || colx >= dem.cols() - 1 || rowx >= dem.rows() - 1)
      continue;

    // Perturb the height down and up
    for (int sign = -1; sign <= 1; sign += 2) {
      for (int height_it = 0; height_it < heightErrEstim->num_height_samples; height_it++) {
        double dh = sign * heightErrEstim->max_height_error
          * double(height_it)/double(heightErrEstim->num_height_samples);

        if (sign == -1) {
          if (dh < heightErrEstim->height_error_vec(colx, rowx)[0]) {
            // We already determined dh can't go as low, so stop here
            break;
          }
        } else if (sign == 1) {
          if (dh > heightErrEstim->height_error_vec(colx, rowx)[1]) {
            break;
          }
        }

        // Determine where to add the dh. Recall that we compute the intensity
        // at (col, row), while perturbing the dem height at (colx, rowx)
        double left_dh = 0, center_dh = 0, right_dh = 0, bottom_dh = 0, top_dh = 0;
        if      (colx == col - 1 && rowx == row    ) left_dh   = dh; 
        else if (colx == col     && rowx == row    ) center_dh = dh; // won't be reached
        else if (colx == col + 1 && rowx == row    ) right_dh  = dh; 
        else if (colx == col     && rowx == row + 1) bottom_dh = dh; 
        else if (colx == col     && rowx == row - 1) top_dh    = dh; 
        
        double left_h   = dem(col - 1, row)     + left_dh;
        double center_h = dem(col,     row)     + center_dh;
        double right_h  = dem(col + 1, row)     + right_dh;
        double bottom_h = dem(col,     row + 1) + bottom_dh;
        double top_h    = dem(col,     row - 1) + top_dh;

        // TODO(oalexan1): Make this into a function to avoid code duplication!
        
        // The xyz position at the center grid point
        Vector2 lonlat = geo.pixel_to_lonlat(Vector2(col, row));
        double h = center_h;
        Vector3 lonlat3 = Vector3(lonlat(0), lonlat(1), h);
        Vector3 base = geo.datum().geodetic_to_cartesian(lonlat3);

        // The xyz position at the left grid point
        lonlat = geo.pixel_to_lonlat(Vector2(col-1, row));
        h = left_h;
        lonlat3 = Vector3(lonlat(0), lonlat(1), h);
        Vector3 left = geo.datum().geodetic_to_cartesian(lonlat3);

        // The xyz position at the right grid point
        lonlat = geo.pixel_to_lonlat(Vector2(col+1, row));
        h = right_h;
        lonlat3 = Vector3(lonlat(0), lonlat(1), h);
        Vector3 right = geo.datum().geodetic_to_cartesian(lonlat3);

        // The xyz position at the bottom grid point
        lonlat = geo.pixel_to_lonlat(Vector2(col, row+1));
        h = bottom_h;
        lonlat3 = Vector3(lonlat(0), lonlat(1), h);
        Vector3 bottom = geo.datum().geodetic_to_cartesian(lonlat3);

        // The xyz position at the top grid point
        lonlat = geo.pixel_to_lonlat(Vector2(col, row-1));
        h = top_h;
        lonlat3 = Vector3(lonlat(0), lonlat(1), h);
        Vector3 top = geo.datum().geodetic_to_cartesian(lonlat3);

        // four-point normal (centered)
        Vector3 dx = right - left;
        Vector3 dy = bottom - top;

        Vector3 normal = -normalize(cross_prod(dx, dy)); // so normal points up

        double phase_angle = 0.0;
        PixelMask<double> reflectance = ComputeReflectance(cameraPosition,
                                                           normal, base, local_model_params,
                                                           global_params, phase_angle,
                                                           reflectance_model_coeffs);
        reflectance.validate();

        double comp_intensity = albedo(col, row) *
          nonlin_reflectance(reflectance, opt.image_exposures_vec[image_iter],
                             opt.steepness_factor,
                             &opt.image_haze_vec[image_iter][0], opt.num_haze_coeffs);

        if (std::abs(comp_intensity - meas_intensity) > max_intensity_err) {
          // We exceeded the error budget, record the dh at which it happens
          if (sign == -1) {
            heightErrEstim->height_error_vec(colx, rowx)[0] = dh;
          } else if (sign == 1) {
            heightErrEstim->height_error_vec(colx, rowx)[1] = dh;
          }
                
          break;
        }

      }
    }
  }
}

bool computeReflectanceAndIntensity(double left_h, double center_h, double right_h,
                                    double bottom_h, double top_h,
                                    bool use_pq, double p, double q, // dem partial derivatives
                                    int col, int row,
                                    ImageView<double>         const& dem,
                                    cartography::GeoReference const& geo,
                                    bool model_shadows,
                                    double max_dem_height,
                                    double gridx, double gridy,
                                    ModelParams  const & model_params,
                                    GlobalParams const & global_params,
                                    BBox2i       const & crop_box,
                                    MaskedImgT   const & image,
                                    DoubleImgT   const & blend_weight,
                                    CameraModel  const * camera,
                                    double       const * scaled_sun_posn,
                                    PixelMask<double>  & reflectance,
                                    PixelMask<double>  & intensity,
                                    double             & ground_weight,
                                    const double       * reflectance_model_coeffs,
                                    SlopeErrEstim      * slopeErrEstim = NULL,
                                    HeightErrEstim     * heightErrEstim = NULL) {

  // Set output values
  reflectance = 0.0; reflectance.invalidate();
  intensity   = 0.0; intensity.invalidate();
  ground_weight = 0.0;
  
  if (col >= dem.cols() - 1 || row >= dem.rows() - 1) return false;
  if (crop_box.empty()) return false;

  if (use_pq) {
    // p is defined as (right_h - left_h)/(2*gridx)
    // so, also, p = (right_h - center_h)/gridx
    // Hence, we get the formulas below in terms of p and q.
    right_h  = center_h + gridx*p;
    left_h   = center_h - gridx*p;
    top_h    = center_h + gridy*q;
    bottom_h = center_h - gridy*q;
  }

  // The xyz position at the center grid point
  Vector2 lonlat = geo.pixel_to_lonlat(Vector2(col, row));
  double h = center_h;
  Vector3 lonlat3 = Vector3(lonlat(0), lonlat(1), h);
  Vector3 base = geo.datum().geodetic_to_cartesian(lonlat3);

  // The xyz position at the left grid point
  lonlat = geo.pixel_to_lonlat(Vector2(col-1, row));
  h = left_h;
  lonlat3 = Vector3(lonlat(0), lonlat(1), h);
  Vector3 left = geo.datum().geodetic_to_cartesian(lonlat3);

  // The xyz position at the right grid point
  lonlat = geo.pixel_to_lonlat(Vector2(col+1, row));
  h = right_h;
  lonlat3 = Vector3(lonlat(0), lonlat(1), h);
  Vector3 right = geo.datum().geodetic_to_cartesian(lonlat3);

  // The xyz position at the bottom grid point
  lonlat = geo.pixel_to_lonlat(Vector2(col, row+1));
  h = bottom_h;
  lonlat3 = Vector3(lonlat(0), lonlat(1), h);
  Vector3 bottom = geo.datum().geodetic_to_cartesian(lonlat3);

  // The xyz position at the top grid point
  lonlat = geo.pixel_to_lonlat(Vector2(col, row-1));
  h = top_h;
  lonlat3 = Vector3(lonlat(0), lonlat(1), h);
  Vector3 top = geo.datum().geodetic_to_cartesian(lonlat3);

  // four-point normal (centered)
  Vector3 dx = right - left;
  Vector3 dy = bottom - top;

  Vector3 normal = -normalize(cross_prod(dx, dy)); // so normal points up

  ModelParams local_model_params = model_params;

  // Update the sun position using the scaled sun position variable 
  for (int it = 0; it < 3; it++) 
    local_model_params.sunPosition[it] = scaled_sun_posn[it] * model_params.sunPosition[it]; 
    
  // Update the camera position for the given pixel (camera position
  // is pixel-dependent for linescan cameras).
  Vector2 pix;
  Vector3 cameraPosition;
  try {
    pix = camera->point_to_pixel(base);
    
    // Need camera center only for Lunar Lambertian
    if (global_params.reflectanceType != LAMBERT)
      cameraPosition = camera->camera_center(pix);
    
  } catch(...){
    reflectance = 0.0; reflectance.invalidate();
    intensity   = 0.0; intensity.invalidate();
    ground_weight = 0.0;
    return false;
  }
  
  double phase_angle = 0.0;
  reflectance = ComputeReflectance(cameraPosition,
                                   normal, base, local_model_params,
                                   global_params, phase_angle,
                                   reflectance_model_coeffs);
  reflectance.validate();


  // Since our image is cropped
  pix -= crop_box.min();

  // Check for out of range
  if (pix[0] < 0 || pix[0] >= image.cols() - 1 || pix[1] < 0 || pix[1] >= image.rows() - 1) {
    reflectance = 0.0; reflectance.invalidate();
    intensity   = 0.0; intensity.invalidate();
    ground_weight = 0.0;
    return false;
  }

  InterpolationView<EdgeExtensionView<MaskedImgT, ConstantEdgeExtension>, BilinearInterpolation>
    interp_image = interpolate(image, BilinearInterpolation(),
                               ConstantEdgeExtension());
  intensity = interp_image(pix[0], pix[1]); // this interpolates

  if (g_blend_weight_is_ground_weight) {
    if (blend_weight.cols() != dem.cols() || blend_weight.rows() != dem.rows()) 
      vw::vw_throw(vw::ArgumentErr() << "Ground weight must have the same size as the DEM.\n");
    ground_weight = blend_weight(col, row);
  } else {
    InterpolationView<EdgeExtensionView<DoubleImgT, ConstantEdgeExtension>, BilinearInterpolation>
      interp_weight = interpolate(blend_weight, BilinearInterpolation(),
                                  ConstantEdgeExtension());
    if (blend_weight.cols() > 0 && blend_weight.rows() > 0) // The weight may not exist
      ground_weight = interp_weight(pix[0], pix[1]); // this interpolates
    else
      ground_weight = 1.0;
  }
  
  // Note that we allow negative reflectance. It will hopefully guide
  // the SfS solution the right way.
  if (!is_valid(intensity)) {
    reflectance = 0.0; reflectance.invalidate();
    intensity   = 0.0; intensity.invalidate();
    ground_weight = 0.0;
    return false;
  }

  if (model_shadows) {
    bool inShadow = asp::isInShadow(col, row, local_model_params.sunPosition,
                                    dem, max_dem_height, gridx, gridy,
                                    geo);

    if (inShadow) {
      // The reflectance is valid, it is just zero
      reflectance = 0;
      reflectance.validate();
    }
  }

  if (slopeErrEstim != NULL && is_valid(intensity) && is_valid(reflectance)) {
    
    int image_iter = slopeErrEstim->image_iter;
    Options & opt = *slopeErrEstim->opt; // alias
    ImageView<double> & albedo = *slopeErrEstim->albedo; // alias
    double comp_intensity = albedo(col, row) *
      nonlin_reflectance(reflectance, opt.image_exposures_vec[image_iter],
                         opt.steepness_factor,
                         &opt.image_haze_vec[image_iter][0], opt.num_haze_coeffs);

    // We use twice the discrepancy between the computed and measured intensity
    // as a measure for how far is overall the computed intensity allowed
    // to diverge from the measured intensity
    double max_intensity_err = 2.0 * std::abs(intensity.child() - comp_intensity);
    estimateSlopeError(cameraPosition,
                       normal, base, local_model_params,
                       global_params,
                       reflectance_model_coeffs,
                       intensity.child(),
                       max_intensity_err,
                       col, row, image_iter,
                       opt, albedo,
                       slopeErrEstim);
  }
  
  if (heightErrEstim != NULL && is_valid(intensity) && is_valid(reflectance)) {
    
    int image_iter = heightErrEstim->image_iter;
    Options & opt = *heightErrEstim->opt; // alias
    ImageView<double> & albedo = *heightErrEstim->albedo; // alias
    double comp_intensity = albedo(col, row) *
      nonlin_reflectance(reflectance, opt.image_exposures_vec[image_iter],
                         opt.steepness_factor,
                         &opt.image_haze_vec[image_iter][0], opt.num_haze_coeffs);
    
    // We use twice the discrepancy between the computed and measured intensity
    // as a measure for how far is overall the computed intensity allowed
    // to diverge from the measured intensity
    double max_intensity_err = 2.0 * std::abs(intensity.child() - comp_intensity);

    estimateHeightError(dem, geo,  
                        cameraPosition, local_model_params,  global_params,  
                        reflectance_model_coeffs, intensity.child(),  
                        max_intensity_err, col, row, image_iter, opt,  albedo,  
                        heightErrEstim);
  }
  
  return true;
}

void computeReflectanceAndIntensity(ImageView<double> const& dem,
                                    ImageView<Vector2> const& pq,
                                    cartography::GeoReference const& geo,
                                    bool model_shadows,
                                    double & max_dem_height, // alias
                                    double gridx, double gridy,
                                    int sample_col_rate, int sample_row_rate,
                                    ModelParams const& model_params,
                                    GlobalParams const& global_params,
                                    BBox2i const& crop_box,
                                    MaskedImgT const  & image,
                                    DoubleImgT const  & blend_weight,
                                    CameraModel const * camera,
                                    double     const  * scaled_sun_posn,
                                    ImageView<PixelMask<double>> & reflectance,
                                    ImageView<PixelMask<double>> & intensity,
                                    ImageView<double>            & ground_weight,
                                    const double   * reflectance_model_coeffs,
                                    SlopeErrEstim  * slopeErrEstim = NULL,
                                    HeightErrEstim * heightErrEstim = NULL) {
  
  // Update max_dem_height
  max_dem_height = -std::numeric_limits<double>::max();
  if (model_shadows) {
    for (int col = 0; col < dem.cols(); col += sample_col_rate) {
      for (int row = 0; row < dem.rows(); row += sample_row_rate) {
        if (dem(col, row) > max_dem_height) {
          max_dem_height = dem(col, row);
        }
      }
    }
    vw_out() << "Maximum DEM height: " << max_dem_height << std::endl;
  }
  
  // Init the reflectance and intensity as invalid. Do it at all grid
  // points, not just where we sample, to ensure that these quantities
  // are fully initialized.
  reflectance.set_size(dem.cols(), dem.rows());
  intensity.set_size(dem.cols(), dem.rows());
  ground_weight.set_size(dem.cols(), dem.rows());
  for (int col = 0; col < dem.cols(); col++) {
    for (int row = 0; row < dem.rows(); row++) {
      reflectance(col, row).invalidate();
      intensity(col, row).invalidate();
      ground_weight(col, row) = 0.0;
    }
  }

  bool use_pq = (pq.cols() > 0 && pq.rows() > 0);
  for (int col = 1; col < dem.cols() - 1; col += sample_col_rate) {
    for (int row = 1; row < dem.rows() - 1; row += sample_row_rate) {
      
      double pval = 0, qval = 0;
      if (use_pq) {
        pval = pq(col, row)[0];
        qval = pq(col, row)[1];
      }
      computeReflectanceAndIntensity(dem(col-1, row), dem(col, row), dem(col+1, row),
                                     dem(col, row+1), dem(col, row-1),
                                     use_pq, pval, qval,
                                     col, row, dem,  geo,
                                     model_shadows, max_dem_height,
                                     gridx, gridy,
                                     model_params, global_params,
                                     crop_box, image, blend_weight, camera,
                                     scaled_sun_posn, 
                                     reflectance(col, row), intensity(col, row),
                                     ground_weight(col, row),
                                     reflectance_model_coeffs,
                                     slopeErrEstim,
                                     heightErrEstim);
    }
  }
  
  return;
}

std::string exposure_file_name(std::string const& prefix){
  return prefix + "-exposures.txt";
}

std::string haze_file_name(std::string const& prefix){
  return prefix + "-haze.txt";
}

std::string model_coeffs_file_name(std::string const& prefix){
  return prefix + "-model_coeffs.txt";
}

// Form a finer resolution image with given dimensions from a coarse image.
// Use constant edge extension.
void interp_image(ImageView<double> const& coarse_image, double scale,
                  ImageView<double> & fine_image){

  ImageViewRef<double> coarse_interp = interpolate(coarse_image,
                                                   BicubicInterpolation(), ConstantEdgeExtension());
  for (int col = 0; col < fine_image.cols(); col++) {
    for (int row = 0; row < fine_image.rows(); row++) {
      fine_image(col, row) = coarse_interp(col*scale, row*scale);
    }
  }
}

void save_exposures(std::string const& out_prefix,
                    std::vector<std::string> const& input_images,
                    std::vector<double> const& exposures){
  std::string exposure_file = exposure_file_name(out_prefix);
  vw_out() << "Writing: " << exposure_file << std::endl;
  std::ofstream exf(exposure_file.c_str());
  exf.precision(18);
  for (size_t image_iter = 0; image_iter < exposures.size(); image_iter++)
    exf << input_images[image_iter] << " " << exposures[image_iter] << "\n";
  exf.close();
}

// Find the sun azimuth and elevation at the lon-lat position of the
// center of the DEM. The result can change depending on the DEM.
void sun_angles(Options const& opt,
                ImageView<double> const& dem, double nodata_val, GeoReference const& georef,
                boost::shared_ptr<CameraModel> cam,
                Vector3 const& sun_pos,
                double & azimuth, double &elevation){

  int cols = dem.cols(), rows = dem.rows();
  if (cols <= 0 || rows <= 0)
    vw_throw( ArgumentErr() << "Expecting a non-empty DEM.\n" );

  // Find lon-lat-height in the middle of the DEM
  Vector2 ll = georef.pixel_to_lonlat(Vector2(cols/2.0, rows/2.0));
  double height = dem(cols/2.0, rows/2.0);
  if (height == nodata_val)
    height = 0;
  Vector3 llh(ll[0], ll[1], height);

  Vector3 xyz = georef.datum().geodetic_to_cartesian(llh); // point on the planet
  Vector3 east(-xyz[1], xyz[0], 0);

  Vector3 sun_dir = sun_pos - xyz;

  //double prod = dot_prod(sun_dir, east)
  //  / sqrt ( dot_prod(east, east) * dot_prod(sun_dir, sun_dir));
  //double angle = (180.0/M_PI) * acos (prod);

  // Projection in the tangent plane
  //  Vector3 proj = sun_dir - xyz*dot_prod(sun_dir, xyz)/dot_prod(xyz, xyz);

  //prod = dot_prod(proj, east)
  //  / sqrt ( dot_prod(east, east) * dot_prod(proj, proj));

  //double angle2 = (180.0/M_PI) * acos (prod);

  // Find the sun direction in the North-East-Down coordinate system
  vw::Matrix3x3 M = georef.datum().lonlat_to_ned_matrix(ll);
  Vector3 sun_dir_ned = inverse(M)*sun_dir;
  
  if (sun_dir_ned[0] == 0 && sun_dir_ned[1] == 0)
    azimuth = 0;
  else
    azimuth = (180.0/M_PI) * atan2(sun_dir_ned[1], sun_dir_ned[0]);

  double L = norm_2(subvector(sun_dir_ned, 0, 2));
  elevation = (180.0/M_PI) * atan2(-sun_dir_ned[2], L);
}

// A function to invoke at every iteration of ceres.
// We need a lot of global variables to do something useful.
Options                               const * g_opt = NULL;
int                                           g_iter = -1;
std::vector<ImageView<double>>              * g_dem = NULL;
std::vector<ImageView<Vector2>>             * g_pq = NULL;
std::vector<ImageView<double>>              * g_albedo = NULL;
std::vector<cartography::GeoReference> const * g_geo = NULL;
GlobalParams                           const * g_global_params = NULL;
std::vector<ModelParams>               const * g_model_params = NULL;
std::vector<std::vector<BBox2i>>       const * g_crop_boxes = NULL;
std::vector<std::vector<MaskedImgT>>   const * g_masked_images = NULL;
std::vector<std::vector<DoubleImgT>>   const * g_blend_weights = NULL;
std::vector<std::vector<boost::shared_ptr<CameraModel>> > * g_cameras = NULL;
double                                       * g_dem_nodata_val = NULL;
float                                        * g_img_nodata_val = NULL;
std::vector<double>                          * g_exposures = NULL;
std::vector<std::vector<double>>             * g_haze = NULL;
std::vector<double>                          * g_adjustments = NULL;
std::vector<double>                          * g_scaled_sun_posns = NULL;
std::vector<double>                          * g_max_dem_height = NULL;
double                                       * g_gridx = NULL;
double                                       * g_gridy = NULL;
int                                            g_level = -1;
bool                                           g_final_iter = false;
double                                       * g_reflectance_model_coeffs = NULL; 

// When floating the camera position and orientation, multiply the
// position variables by this factor times
// opt.camera_position_step_size to give it a greater range in
// searching (it makes more sense to wiggle the camera position by say
// 1 meter than by a tiny fraction of one millimeter).
double g_position_scale_factor = 1e+6;

class SfsCallback: public ceres::IterationCallback {
public:
  virtual ceres::CallbackReturnType operator()
    (const ceres::IterationSummary& summary) {

    g_iter++;

    vw_out() << "Finished iteration: " << g_iter << std::endl;
    // callTop();

    if (!g_opt->save_computed_intensity_only)
      save_exposures(g_opt->out_prefix, g_opt->input_images, *g_exposures);

    if (g_opt->num_haze_coeffs > 0 && !g_opt->save_computed_intensity_only) {
      std::string haze_file = haze_file_name(g_opt->out_prefix);
      vw_out() << "Writing: " << haze_file << std::endl;
      std::ofstream hzf(haze_file.c_str());
      hzf.precision(18);
      for (size_t image_iter = 0; image_iter < (*g_haze).size(); image_iter++) {
        hzf << g_opt->input_images[image_iter];
        for (size_t hiter = 0; hiter < (*g_haze)[image_iter].size(); hiter++) {
          hzf << " " << (*g_haze)[image_iter][hiter];
        }
        hzf << "\n";
      }
      hzf.close();
    }
    
    std::string model_coeffs_file = model_coeffs_file_name(g_opt->out_prefix);
    if (!g_opt->save_computed_intensity_only) {
      vw_out() << "Writing: " << model_coeffs_file << std::endl;
      std::ofstream mcf(model_coeffs_file.c_str());
      mcf.precision(18);
      for (size_t coeff_iter = 0; coeff_iter < g_num_model_coeffs; coeff_iter++){
        mcf << g_reflectance_model_coeffs[coeff_iter] << " ";
      }
      mcf << "\n";
      mcf.close();
    }
    
    //vw_out() << "Model coefficients: "; 
    //for (size_t i = 0; i < g_num_model_coeffs; i++)
    //  vw_out() << g_reflectance_model_coeffs[i] << " ";
    //vw_out() << std::endl;
    
    //if (!g_opt->use_approx_adjusted_camera_models) {
    //  vw_out() << "cam adj: ";
    //  for (int s = 0; s < int((*g_adjustments).size()); s++) {
    //    vw_out() << (*g_adjustments)[s] << " ";
    // }
    // vw_out() << std::endl;
    //}
    
    //vw_out() << "scaled sun position: ";
    //for (int s = 0; s < int((*g_scaled_sun_posns).size()); s++) 
    //  vw_out() << (*g_scaled_sun_posns)[s] << " ";
    //vw_out() << std::endl;

    int num_dems = (*g_dem).size();
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      
      // Apply the most recent adjustments to the cameras.
      for (size_t image_iter = 0; image_iter < (*g_masked_images)[dem_iter].size(); image_iter++) {
        if (g_opt->skip_images[dem_iter].find(image_iter) !=
            g_opt->skip_images[dem_iter].end()) continue;

        if (!g_opt->use_approx_adjusted_camera_models) {
          // When we use approx adjusted camera models we never change the adjustments,
          // hence this step is not necessary
          AdjustedCameraModel * icam
            = dynamic_cast<AdjustedCameraModel*>((*g_cameras)[dem_iter][image_iter].get());
          if (icam == NULL)
            vw_throw(ArgumentErr() << "Expecting an adjusted camera model.\n");
          Vector3 translation;
          Vector3 axis_angle;
          for (int param_iter = 0; param_iter < 3; param_iter++) {
            translation[param_iter]
              = (g_position_scale_factor*g_opt->camera_position_step_size)*
              (*g_adjustments)[6*image_iter + 0 + param_iter];
            axis_angle[param_iter] = (*g_adjustments)[6*image_iter + 3 + param_iter];
          }
          icam->set_translation(translation);
          icam->set_axis_angle_rotation(axis_angle);
        }
      }

      std::ostringstream os;
      if (!g_final_iter) {
        os << "-iter" << g_iter;
      }else{
        os << "-final";
      }

      // Note that for level 0 we don't append the level as part of
      // the filename. This way, whether we have levels or not,
      // the lowest level is always named consistently.
      if ((*g_opt).coarse_levels > 0 && g_level > 0) os << "-level" << g_level;
      if (num_dems > 1)                              os << "-clip"  << dem_iter;

      std::string iter_str = os.str();

      // The DEM with no-data where there are no valid image pixels
      ImageView<double> dem_nodata;
      if (g_opt->save_dem_with_nodata) {
        dem_nodata = ImageView<double>((*g_dem)[dem_iter].cols(), (*g_dem)[dem_iter].rows());
        fill(dem_nodata, *g_dem_nodata_val);
      }
        
      bool has_georef = true, has_nodata = true;
      TerminalProgressCallback tpc("asp", ": ");
      if ( (!g_opt->save_sparingly || g_final_iter) && !g_opt->save_computed_intensity_only ) {
        std::string out_dem_file = g_opt->out_prefix + "-DEM"
          + iter_str + ".tif";
        vw_out() << "Writing: " << out_dem_file << std::endl;
        block_write_gdal_image(out_dem_file, (*g_dem)[dem_iter], has_georef, (*g_geo)[dem_iter],
                               has_nodata, *g_dem_nodata_val,
                               *g_opt, tpc);
      }
      
      if ((!g_opt->save_sparingly || (g_final_iter && g_opt->float_albedo)) &&
          !g_opt->save_computed_intensity_only ) {
        std::string out_albedo_file = g_opt->out_prefix + "-comp-albedo"
          + iter_str + ".tif";
        vw_out() << "Writing: " << out_albedo_file << std::endl;
        block_write_gdal_image(out_albedo_file, (*g_albedo)[dem_iter], has_georef,
                               (*g_geo)[dem_iter],
                               has_nodata, *g_dem_nodata_val,
                               *g_opt, tpc);
      }

      // Print reflectance and other things
      for (size_t image_iter = 0; image_iter < (*g_masked_images)[dem_iter].size(); image_iter++) {

        if (g_opt->skip_images[dem_iter].find(image_iter) !=
            g_opt->skip_images[dem_iter].end()) {
          continue;
        }
      
        ImageView<PixelMask<double>> reflectance, intensity, comp_intensity;
        ImageView<double> ground_weight;

        std::string out_camera_file
          = asp::bundle_adjust_file_name(g_opt->out_prefix,
                                         g_opt->input_images[image_iter],
                                         g_opt->input_cameras[image_iter]);
        
        if (!g_opt->use_approx_adjusted_camera_models) {
          // Save the camera adjustments for the current iteration.
          // When we use approx adjusted camera models this is not necessary
          // since then the cameras don't change.
          //std::string out_camera_file = g_opt->out_prefix + "-camera"
          //+ iter_str2 + ".adjust";
          //vw_out() << "Writing: " << out_camera_file << std::endl;
          AdjustedCameraModel * icam
            = dynamic_cast<AdjustedCameraModel*>((*g_cameras)[dem_iter][image_iter].get());
          if (icam == NULL)
            vw_throw( ArgumentErr() << "Expecting an adjusted camera.\n");
          Vector3 translation = icam->translation();
          Quaternion<double> rotation = icam->rotation();
          //asp::write_adjustments(out_camera_file, translation, rotation);
          
          // Used to save adjusted files in the format <out prefix>-<input-img>.adjust
          // so we can later read them with --bundle-adjust-prefix to be
          // used in another SfS run. Don't do that anymore as normally
          // the adjustments don't change.
          //if (g_level == 0) {
          //if (!g_opt->save_computed_intensity_only){
          //  vw_out() << "Writing: " << out_camera_file << std::endl;
          //  asp::write_adjustments(out_camera_file, translation, rotation);
          //}
        }
        
        if (g_opt->save_sparingly && !g_opt->save_dem_with_nodata) 
          continue; // don't write too many things
        
        // Manufacture an output prefix for the other data associated with this camera
        std::string iter_str2 = fs::path(out_camera_file).replace_extension("").string();
        iter_str2 += iter_str;
    
        // Compute reflectance and intensity with optimized DEM
        int sample_col_rate = 1, sample_row_rate = 1;
        computeReflectanceAndIntensity((*g_dem)[dem_iter], (*g_pq)[dem_iter],
                                       (*g_geo)[dem_iter],
                                       g_opt->model_shadows,
                                       (*g_max_dem_height)[dem_iter],
                                       *g_gridx, *g_gridy,
                                       sample_col_rate, sample_row_rate,
                                       (*g_model_params)[image_iter],
                                       *g_global_params,
                                       (*g_crop_boxes)[dem_iter][image_iter],
                                       (*g_masked_images)[dem_iter][image_iter],
                                       (*g_blend_weights)[dem_iter][image_iter],
                                       (*g_cameras)[dem_iter][image_iter].get(),
                                       &(*g_scaled_sun_posns)[3*image_iter],
                                       reflectance, intensity, ground_weight, 
                                       g_reflectance_model_coeffs);

        // dem_nodata equals to dem if the image has valid pixels and no shadows
        if (g_opt->save_dem_with_nodata) {
          for (int col = 0; col < reflectance.cols(); col++) {
            for (int row = 0; row < reflectance.rows(); row++) {
              if (is_valid(reflectance(col, row))) 
                dem_nodata(col, row) = (*g_dem)[dem_iter](col, row);
            }
          }
        }

        if (g_opt->save_sparingly)
          continue;
        
        // Find the computed intensity
        comp_intensity.set_size(reflectance.cols(), reflectance.rows());
        for (int col = 0; col < comp_intensity.cols(); col++) {
          for (int row = 0; row < comp_intensity.rows(); row++) {
            comp_intensity(col, row)
              = (*g_albedo)[dem_iter](col, row) *
              nonlin_reflectance(reflectance(col, row), (*g_exposures)[image_iter],
                                 g_opt->steepness_factor,
                                 &(*g_haze)[image_iter][0], g_opt->num_haze_coeffs);
          }
        }

        std::string out_meas_intensity_file = iter_str2 + "-meas-intensity.tif";
        vw_out() << "Writing: " << out_meas_intensity_file << std::endl;
        block_write_gdal_image(out_meas_intensity_file,
                               apply_mask(intensity, *g_img_nodata_val),
                               has_georef, (*g_geo)[dem_iter], has_nodata,
                               *g_img_nodata_val, *g_opt, tpc);
    
        std::string out_comp_intensity_file = iter_str2 + "-comp-intensity.tif";
        vw_out() << "Writing: " << out_comp_intensity_file << std::endl;
        block_write_gdal_image(out_comp_intensity_file,
                               apply_mask(comp_intensity, *g_img_nodata_val),
                               has_georef, (*g_geo)[dem_iter], has_nodata, *g_img_nodata_val,
                               *g_opt, tpc);

        if (g_opt->save_computed_intensity_only) 
          continue; // don't write too many things

        std::string out_weight_file = iter_str2 + "-blending-weight.tif";
        vw_out() << "Writing: " << out_weight_file << std::endl;
        block_write_gdal_image(out_weight_file,
                               ground_weight,
                               has_georef, (*g_geo)[dem_iter], has_nodata, *g_img_nodata_val,
                               *g_opt, tpc);

        std::string out_reflectance_file = iter_str2 + "-reflectance.tif";
        vw_out() << "Writing: " << out_reflectance_file << std::endl;
        block_write_gdal_image(out_reflectance_file,
                               apply_mask(reflectance, *g_img_nodata_val),
                               has_georef, (*g_geo)[dem_iter], has_nodata, *g_img_nodata_val,
                               *g_opt, tpc);


        // Find the measured normalized albedo, after correcting for
        // reflectance.
        ImageView<double> measured_albedo;
        measured_albedo.set_size(reflectance.cols(), reflectance.rows());
        for (int col = 0; col < measured_albedo.cols(); col++) {
          for (int row = 0; row < measured_albedo.rows(); row++) {
            if (!is_valid(reflectance(col, row)))
              measured_albedo(col, row) = 1;
            else
              measured_albedo(col, row) = intensity(col, row) /
                nonlin_reflectance(reflectance(col, row), (*g_exposures)[image_iter],
                                   g_opt->steepness_factor,
                                   &(*g_haze)[image_iter][0], g_opt->num_haze_coeffs);
          }
        }
        std::string out_albedo_file = iter_str2 + "-meas-albedo.tif";
        vw_out() << "Writing: " << out_albedo_file << std::endl;
        block_write_gdal_image(out_albedo_file, measured_albedo,
                               has_georef, (*g_geo)[dem_iter], has_nodata, 0, *g_opt, tpc);


        double imgmean, imgstdev, refmean, refstdev;
        compute_image_stats(intensity, comp_intensity, imgmean, imgstdev, refmean, refstdev);
        vw_out() << "meas image mean and std: " << imgmean << ' ' << imgstdev
                 << std::endl;
        vw_out() << "comp image mean and std: " << refmean << ' ' << refstdev
                 << std::endl;

        vw_out() << "Exposure for image " << image_iter << ": "
                 << (*g_exposures)[image_iter] << std::endl;

        if (g_opt->num_haze_coeffs > 0) {
          vw_out() << "Haze for image " << image_iter << ":";
          for (size_t hiter = 0; hiter < (*g_haze)[image_iter].size(); hiter++) {
            vw_out() << " " << (*g_haze)[image_iter][hiter];
          }
          vw_out() << std::endl;
        }
      }

      if (g_opt->save_dem_with_nodata) {
        if ( !g_opt->save_sparingly || g_final_iter ) {
          std::string out_dem_nodata_file = g_opt->out_prefix + "-DEM-nodata"
            + iter_str + ".tif";
          vw_out() << "Writing: " << out_dem_nodata_file << std::endl;
          TerminalProgressCallback tpc("asp", ": ");
          block_write_gdal_image(out_dem_nodata_file, dem_nodata,
                                 has_georef, (*g_geo)[dem_iter],
                                 has_nodata, *g_dem_nodata_val,
                                 *g_opt, tpc);
        }
      }
    }
    
    return ceres::SOLVER_CONTINUE;
  }
};

// See SmoothnessError() for the definitions of bottom, top, etc.
template <typename F, typename G>
inline bool
calc_intensity_residual(const F* const exposure,
                        const F* const haze,
                        const G* const left,
                        const G* const center,
                        const G* const right,
                        const G* const bottom,
                        const G* const top,
                        bool use_pq,
                        const G* const pq, // partial derivatives of the dem in x and y
                        const G* const albedo,
                        const F* const camera_adjustments,
                        const F* const scaled_sun_posn,
                        const G* const reflectance_model_coeffs, 
                        int m_col, int m_row,
                        ImageView<double>                 const & m_dem,            // alias
                        cartography::GeoReference         const & m_geo,            // alias
                        bool                                      m_model_shadows,
                        double                                    m_camera_position_step_size,
                        double                            const & m_max_dem_height, // alias
                        double                                    m_gridx,
                        double                                    m_gridy,
                        GlobalParams                      const & m_global_params,  // alias
                        ModelParams                       const & m_model_params,   // alias
                        BBox2i                                    m_crop_box,
                        MaskedImgT                        const & m_image,          // alias
                        DoubleImgT                        const & m_blend_weight,   // alias
                        boost::shared_ptr<CameraModel>    const & m_camera,         // alias
                        F* residuals) {
  
  // Default residuals. Using here 0 rather than some big number tuned out to
  // work better than the alternative.
  residuals[0] = F(0.0);
  try{

    // Initialize this variable to something for now, it does not
    // matter yet to what.  We just don't want it to go out of scope.
    AdjustedCameraModel adj_cam_copy(m_camera);

    // This is a bit tricky. If use adjusted approximate camera model
    // (then the cameras never change), use the camera passed
    // in. Else, create a copy of this camera to avoid issues when
    // using multiple threads. In that case we copy just the
    // adjustment parameters, the pointer to the underlying ISIS
    // camera is shared.
    CameraModel * camera = NULL;

    if (g_opt->use_approx_adjusted_camera_models) {
      camera = (CameraModel*)(m_camera.get());
    }else{
      AdjustedCameraModel * adj_cam
        = dynamic_cast<AdjustedCameraModel*>(m_camera.get());
      if (adj_cam == NULL)
        vw_throw( ArgumentErr() << "Expecting an adjusted camera.\n");

      // We overwrite the dummy value of adj_cam_copy with a deep copy
      // of adj_cam.
      adj_cam_copy = *adj_cam;
      
      // Apply current adjustments to the camera
      Vector3 axis_angle;
      Vector3 translation;
      for (int param_iter = 0; param_iter < 3; param_iter++) {
        translation[param_iter]
          = (g_position_scale_factor*m_camera_position_step_size)*camera_adjustments[param_iter];
        axis_angle[param_iter] = camera_adjustments[3 + param_iter];
      }
      adj_cam_copy.set_translation(translation);
      adj_cam_copy.set_axis_angle_rotation(axis_angle);
      
      camera = &adj_cam_copy;
    }
    
    PixelMask<double> reflectance(0), intensity(0);
    double ground_weight = 0;

    // Need to be careful not to access an array which does not exist
    G p = 0, q = 0;
    if (use_pq) {
      p = pq[0];
      q = pq[1];
    }
    
    bool success =
      computeReflectanceAndIntensity(left[0], center[0], right[0],
                                     bottom[0], top[0],
                                     use_pq, p, q,
                                     m_col, m_row,  m_dem, m_geo,
                                     m_model_shadows, m_max_dem_height,
                                     m_gridx, m_gridy,
                                     m_model_params,  m_global_params,
                                     m_crop_box, m_image, m_blend_weight, camera,
                                     scaled_sun_posn,
                                     reflectance, intensity, ground_weight, reflectance_model_coeffs);
      
    if (g_opt->unreliable_intensity_threshold > 0){
      if (is_valid(intensity) && intensity.child() <= g_opt->unreliable_intensity_threshold &&
          intensity.child() >= 0) {
        ground_weight *=
          pow(intensity.child()/g_opt->unreliable_intensity_threshold, 2.0);
      }
    }
      
    if (success && is_valid(intensity) && is_valid(reflectance))
      residuals[0] = ground_weight * (intensity - albedo[0] *
                               nonlin_reflectance(reflectance.child(), exposure[0],
                                                  g_opt->steepness_factor,
                                                  haze, g_opt->num_haze_coeffs));
    

  } catch (const camera::PointToPixelErr& e) {
    // To be able to handle robustly DEMs that extend beyond the camera,
    // always return true when we fail to project, but with zero residual.
    // This needs more study.
    residuals[0] = F(0.0);
    return true;
  }

  return true;
}

// Discrepancy between measured and computed intensity.
// sum_i | I_i - albedo * nonlin_reflectance(reflectance_i, exposures[i], haze, num_haze_coeffs) |^2
struct IntensityError {
  IntensityError(int col, int row,
                 ImageView<double> const& dem,
                 cartography::GeoReference const& geo,
                 bool model_shadows,
                 double camera_position_step_size,
                 double const& max_dem_height, // note: this is an alias
                 double gridx, double gridy,
                 GlobalParams const& global_params,
                 ModelParams const& model_params,
                 BBox2i const& crop_box,
                 MaskedImgT const& image,
                 DoubleImgT const& blend_weight,
                 double * scaled_sun_posn, 
                 boost::shared_ptr<CameraModel> const& camera):
    m_col(col), m_row(row), m_dem(dem), m_geo(geo),
    m_model_shadows(model_shadows),
    m_camera_position_step_size(camera_position_step_size),
    m_max_dem_height(max_dem_height),
    m_gridx(gridx), m_gridy(gridy),
    m_global_params(global_params),
    m_model_params(model_params),
    m_crop_box(crop_box),
    m_image(image), m_blend_weight(blend_weight),
    m_scaled_sun_posn(scaled_sun_posn),
    m_camera(camera) {}

  // See SmoothnessError() for the definitions of bottom, top, etc.
  template <typename F>
  bool operator()(const F* const exposure,
                  const F* const haze,
                  const F* const left,
                  const F* const center,
                  const F* const right,
                  const F* const bottom,
                  const F* const top,
                  const F* const albedo,
                  const F* const camera_adjustments,
                  //const F* const scaled_sun_posn,
                  const F* const reflectance_model_coeffs,
                  F* residuals) const {

    // For this error we do not use p and q, hence just use a placeholder.
    bool use_pq = false;
    const F * const pq = NULL;

    //const F* const haze = NULL;
    return calc_intensity_residual(exposure, haze,
                                   left, center, right, bottom, top,
                                   use_pq, pq,
                                   albedo, camera_adjustments,
                                   m_scaled_sun_posn,
                                   reflectance_model_coeffs,
                                   m_col, m_row,  
                                   m_dem,  // alias
                                   m_geo,  // alias
                                   m_model_shadows,  
                                   m_camera_position_step_size,  
                                   m_max_dem_height,  // alias
                                   m_gridx, m_gridy,  
                                   m_global_params,   // alias
                                   m_model_params,    // alias
                                   m_crop_box,  
                                   m_image,           // alias
                                   m_blend_weight,    // alias
                                   m_camera,          // alias
                                   residuals);
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(int col, int row,
                                     ImageView<double> const& dem,
                                     vw::cartography::GeoReference const& geo,
                                     bool model_shadows,
                                     double camera_position_step_size,
                                     double const& max_dem_height, // alias
                                     double gridx, double gridy,
                                     GlobalParams const& global_params,
                                     ModelParams const& model_params,
                                     BBox2i const& crop_box,
                                     MaskedImgT const& image,
                                     DoubleImgT const& blend_weight,
                                     double * scaled_sun_posn, 
                                     boost::shared_ptr<CameraModel> const& camera){
    return (new ceres::NumericDiffCostFunction<IntensityError,
            ceres::CENTRAL, 1, 1, g_max_num_haze_coeffs, 1, 1, 1, 1, 1, 1, 6, g_num_model_coeffs>
            (new IntensityError(col, row, dem, geo,
                                model_shadows,
                                camera_position_step_size,
                                max_dem_height,
                                gridx, gridy,
                                global_params, model_params,
                                crop_box, image, blend_weight, scaled_sun_posn, camera)));
  }

  int m_col, m_row;
  ImageView<double>                 const & m_dem;            // alias
  cartography::GeoReference         const & m_geo;            // alias
  bool                                      m_model_shadows;
  double                                    m_camera_position_step_size;
  double                            const & m_max_dem_height; // alias
  double                                    m_gridx, m_gridy;
  GlobalParams                      const & m_global_params;  // alias
  ModelParams                       const & m_model_params;   // alias
  BBox2i                                    m_crop_box;
  MaskedImgT                        const & m_image;          // alias
  DoubleImgT                        const & m_blend_weight;   // alias
  double                                  * m_scaled_sun_posn;   //  pointer
  boost::shared_ptr<CameraModel>    const & m_camera;         // alias
};

// A variation of the intensity error where only the DEM is floated
struct IntensityErrorFloatDemOnly {
  IntensityErrorFloatDemOnly(int col, int row,
                             ImageView<double> const& dem,
                             double albedo,
                             double * reflectance_model_coeffs, 
                             double * exposure, 
                             double * haze, 
                             double * camera_adjustments, 
                             cartography::GeoReference const& geo,
                             bool model_shadows,
                             double camera_position_step_size,
                             double const& max_dem_height, // note: this is an alias
                             double gridx, double gridy,
                             GlobalParams const& global_params,
                             ModelParams const& model_params,
                             BBox2i const& crop_box,
                             MaskedImgT const& image,
                             DoubleImgT const& blend_weight,
                             double * scaled_sun_posn, 
                             boost::shared_ptr<CameraModel> const& camera):
    m_col(col), m_row(row), m_dem(dem),
    m_albedo(albedo), m_reflectance_model_coeffs(reflectance_model_coeffs),
    m_exposure(exposure), m_haze(haze), m_camera_adjustments(camera_adjustments),
    m_geo(geo),
    m_model_shadows(model_shadows),
    m_camera_position_step_size(camera_position_step_size),
    m_max_dem_height(max_dem_height),
    m_gridx(gridx), m_gridy(gridy),
    m_global_params(global_params),
    m_model_params(model_params),
    m_crop_box(crop_box),
    m_image(image), m_blend_weight(blend_weight),
    m_scaled_sun_posn(scaled_sun_posn),
    m_camera(camera) {}

  // See SmoothnessError() for the definitions of bottom, top, etc.
  template <typename F>
  bool operator()(const F* const left,
                  const F* const center,
                  const F* const right,
                  const F* const bottom,
                  const F* const top,
                  F* residuals) const {

    // For this error we do not use p and q, hence just use a placeholder.
    bool use_pq = false;
    const F * const pq = NULL;

    return calc_intensity_residual(m_exposure, m_haze,
                                   left, center, right, bottom, top,
                                   use_pq, pq,
                                   &m_albedo, m_camera_adjustments,
                                   m_scaled_sun_posn,
                                   m_reflectance_model_coeffs,
                                   m_col, m_row,  
                                   m_dem,  // alias
                                   m_geo,  // alias
                                   m_model_shadows,  
                                   m_camera_position_step_size,  
                                   m_max_dem_height,  // alias
                                   m_gridx, m_gridy,  
                                   m_global_params,   // alias
                                   m_model_params,    // alias
                                   m_crop_box,  
                                   m_image,           // alias
                                   m_blend_weight,    // alias
                                   m_camera,          // alias
                                   residuals);
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(int col, int row,
                                     ImageView<double> const& dem,
                                     double albedo,
                                     double * reflectance_model_coeffs, 
                                     double * exposure, 
                                     double * haze, 
                                     double * camera_adjustments, 
                                     vw::cartography::GeoReference const& geo,
                                     bool model_shadows,
                                     double camera_position_step_size,
                                     double const& max_dem_height, // alias
                                     double gridx, double gridy,
                                     GlobalParams const& global_params,
                                     ModelParams const& model_params,
                                     BBox2i const& crop_box,
                                     MaskedImgT const& image,
                                     DoubleImgT const& blend_weight,
                                     double * scaled_sun_posn, 
                                     boost::shared_ptr<CameraModel> const& camera){
    return (new ceres::NumericDiffCostFunction<IntensityErrorFloatDemOnly,
            ceres::CENTRAL, 1, 1, 1, 1, 1, 1>
            (new IntensityErrorFloatDemOnly(col, row, dem,
                                            albedo, reflectance_model_coeffs,
                                            exposure, haze, camera_adjustments,
                                            geo,
                                            model_shadows,
                                            camera_position_step_size,
                                            max_dem_height,
                                            gridx, gridy,
                                            global_params, model_params,
                                            crop_box, image, blend_weight, scaled_sun_posn,
                                            camera)));
  }

  int                                       m_col, m_row;
  ImageView<double>                 const & m_dem;            // alias
  double                                    m_albedo;
  double                                  * m_reflectance_model_coeffs;
  double                                  * m_exposure;
  double                                  * m_haze;
  double                                  * m_camera_adjustments;
  cartography::GeoReference         const & m_geo;            // alias
  bool                                      m_model_shadows;
  double                                    m_camera_position_step_size;
  double                            const & m_max_dem_height; // alias
  double                                    m_gridx, m_gridy;
  GlobalParams                      const & m_global_params;  // alias
  ModelParams                       const & m_model_params;   // alias
  BBox2i                                    m_crop_box;
  MaskedImgT                        const & m_image;          // alias
  DoubleImgT                        const & m_blend_weight;   // alias
  double                                  * m_scaled_sun_posn;   // pointer
  boost::shared_ptr<CameraModel>    const & m_camera;         // alias
};

// A variation of IntensityError where albedo, dem, and model params are fixed.
struct IntensityErrorFixedMost {
  IntensityErrorFixedMost(int col, int row,
                          ImageView<double> const& dem,
                          double albedo,
                          double * reflectance_model_coeffs, 
                          cartography::GeoReference const& geo,
                          bool model_shadows,
                          double camera_position_step_size,
                          double const& max_dem_height, // note: this is an alias
                          double gridx, double gridy,
                          GlobalParams const& global_params,
                          ModelParams const& model_params,
                          BBox2i const& crop_box,
                          MaskedImgT const& image,
                          DoubleImgT const& blend_weight,
                          boost::shared_ptr<CameraModel> const& camera):
    m_col(col), m_row(row), m_dem(dem),
    m_albedo(albedo), m_reflectance_model_coeffs(reflectance_model_coeffs), 
    m_geo(geo),
    m_model_shadows(model_shadows),
    m_camera_position_step_size(camera_position_step_size),
    m_max_dem_height(max_dem_height),
    m_gridx(gridx), m_gridy(gridy),
    m_global_params(global_params),
    m_model_params(model_params),
    m_crop_box(crop_box),
    m_image(image), m_blend_weight(blend_weight),
    m_camera(camera) {}

  // See SmoothnessError() for the definitions of bottom, top, etc.
  template <typename F>
  bool operator()(const F* const exposure,
                  const F* const haze,
                  const F* const camera_adjustments,
                  const F* const scaled_sun_posn,
                  F* residuals) const {

    // For this error we do not use p and q, hence just use a placeholder.
    bool use_pq = false;
    const F * const pq = NULL;
    
    return calc_intensity_residual(exposure, haze,
                                   &m_dem(m_col-1, m_row),            // left
                                   &m_dem(m_col, m_row),              // center
                                   &m_dem(m_col+1, m_row),            // right
                                   &m_dem(m_col, m_row+1),            // bottom
                                   &m_dem(m_col, m_row-1),            // top
                                   use_pq, pq,
                                   &m_albedo,
                                   camera_adjustments,
                                   scaled_sun_posn,
                                   m_reflectance_model_coeffs,
                                   m_col, m_row,  
                                   m_dem,  // alias
                                   m_geo,  // alias
                                   m_model_shadows,  
                                   m_camera_position_step_size,  
                                   m_max_dem_height,  // alias
                                   m_gridx, m_gridy,  
                                   m_global_params,  // alias
                                   m_model_params,  // alias
                                   m_crop_box,  
                                   m_image,  // alias
                                   m_blend_weight,  // alias
                                   m_camera,  // alias
                                   residuals);
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(int col, int row,
                                     ImageView<double> const& dem,
                                     double albedo,
                                     double * reflectance_model_coeffs, 
                                     vw::cartography::GeoReference const& geo,
                                     bool model_shadows,
                                     double camera_position_step_size,
                                     double const& max_dem_height, // alias
                                     double gridx, double gridy,
                                     GlobalParams const& global_params,
                                     ModelParams const& model_params,
                                     BBox2i const& crop_box,
                                     MaskedImgT const& image,
                                     DoubleImgT const& blend_weight,
                                     boost::shared_ptr<CameraModel> const& camera){
    return (new ceres::NumericDiffCostFunction<IntensityErrorFixedMost,
            ceres::CENTRAL, 1, 1, g_max_num_haze_coeffs, 6, 3>
            (new IntensityErrorFixedMost(col, row, dem, albedo, reflectance_model_coeffs, geo,
                                         model_shadows,
                                         camera_position_step_size,
                                         max_dem_height,
                                         gridx, gridy,
                                         global_params, model_params,
                                         crop_box, image, blend_weight, camera)));
  }

  int m_col, m_row;
  ImageView<double>                 const & m_dem;            // alias
  double                                    m_albedo;
  double                                  * m_reflectance_model_coeffs; 
  cartography::GeoReference         const & m_geo;            // alias
  bool                                      m_model_shadows;
  double                                    m_camera_position_step_size;
  double                            const & m_max_dem_height; // alias
  double                                    m_gridx, m_gridy;
  GlobalParams                      const & m_global_params;  // alias
  ModelParams                       const & m_model_params;   // alias
  BBox2i                                    m_crop_box;
  MaskedImgT                        const & m_image;          // alias
  DoubleImgT                        const & m_blend_weight;   // alias
  boost::shared_ptr<CameraModel>    const & m_camera;         // alias
};

// A variant of the intensity error when we float the partial derviatives
// in x and in y of the dem, which we call p and q.  
struct IntensityErrorPQ {
  IntensityErrorPQ(int col, int row,
                   ImageView<double> const& dem,
                   cartography::GeoReference const& geo,
                   bool model_shadows,
                   double camera_position_step_size,
                   double const& max_dem_height, // note: this is an alias
                   double gridx, double gridy,
                   GlobalParams const& global_params,
                   ModelParams const& model_params,
                   BBox2i const& crop_box,
                   MaskedImgT const& image,
                   DoubleImgT const& blend_weight,
                   boost::shared_ptr<CameraModel> const& camera):
    m_col(col), m_row(row), m_dem(dem), m_geo(geo),
    m_model_shadows(model_shadows),
    m_camera_position_step_size(camera_position_step_size),
    m_max_dem_height(max_dem_height),
    m_gridx(gridx), m_gridy(gridy),
    m_global_params(global_params),
    m_model_params(model_params),
    m_crop_box(crop_box),
    m_image(image), m_blend_weight(blend_weight),
    m_camera(camera) {}
  
  // See SmoothnessError() for the definitions of bottom, top, etc.
  template <typename F>
  bool operator()(const F* const exposure,
                  const F* const haze,                  
                  const F* const center_h,
                  const F* const pq,                 // array of length 2 
                  const F* const albedo,
                  const F* const camera_adjustments, // array of length 6
                  const F* const scaled_sun_posn,       // array of length 3
                  const F* const reflectance_model_coeffs,
                  F* residuals) const {

    bool use_pq = true;

    F v = 0;
    return calc_intensity_residual(exposure, haze, &v, center_h, &v, &v, &v,
                                   use_pq, pq,
                                   albedo, camera_adjustments,
                                   scaled_sun_posn,
                                   reflectance_model_coeffs,
                                   m_col, m_row,  
                                   m_dem,  // alias
                                   m_geo,  // alias
                                   m_model_shadows,  
                                   m_camera_position_step_size,  
                                   m_max_dem_height,  // alias
                                   m_gridx, m_gridy,  
                                   m_global_params,   // alias
                                   m_model_params,    // alias
                                   m_crop_box,  
                                   m_image,           // alias
                                   m_blend_weight,    // alias
                                   m_camera,          // alias
                                   residuals);
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(int col, int row,
                                     ImageView<double> const& dem,
                                     vw::cartography::GeoReference const& geo,
                                     bool model_shadows,
                                     double camera_position_step_size,
                                     double const& max_dem_height, // alias
                                     double gridx, double gridy,
                                     GlobalParams const& global_params,
                                     ModelParams const& model_params,
                                     BBox2i const& crop_box,
                                     MaskedImgT const& image,
                                     DoubleImgT const& blend_weight,
                                     boost::shared_ptr<CameraModel> const& camera){
    return (new ceres::NumericDiffCostFunction<IntensityErrorPQ,
            ceres::CENTRAL, 1, 1, g_max_num_haze_coeffs, 1, 2, 1, 6, 3, g_num_model_coeffs>
            (new IntensityErrorPQ(col, row, dem, geo,
                                  model_shadows,
                                  camera_position_step_size,
                                  max_dem_height,
                                  gridx, gridy,
                                  global_params, model_params,
                                  crop_box, image, blend_weight, camera)));
  }

  int m_col, m_row;
  ImageView<double>                 const & m_dem;            // alias
  cartography::GeoReference         const & m_geo;            // alias
  bool                                      m_model_shadows;
  double                                    m_camera_position_step_size;
  double                            const & m_max_dem_height; // alias
  double                                    m_gridx, m_gridy;
  GlobalParams                      const & m_global_params;  // alias
  ModelParams                       const & m_model_params;   // alias
  BBox2i                                    m_crop_box;
  MaskedImgT                        const & m_image;          // alias
  DoubleImgT                        const & m_blend_weight;   // alias
  boost::shared_ptr<CameraModel>    const & m_camera;         // alias
};

// The smoothness error is the sum of squares of
// the 4 second order partial derivatives, with a weight:
// error = smoothness_weight * ( u_xx^2 + u_xy^2 + u_yx^2 + u_yy^2 )

// We will use finite differences to compute these.
// Consider a grid point and its neighbors, 9 points in all.
//
// bl   = u(c-1, r+1)  bottom = u(c, r+1) br    = u(c+1,r+1)
// left = u(c-1, r  )  center = u(c, r  ) right = u(c+1,r  )
// tl   = u(c-1, r-1)  top    = u(c, r-1) tr    = u(c+1,r-1)
//
// See https://en.wikipedia.org/wiki/Finite_difference
// for the obtained formulas.

struct SmoothnessError {
  SmoothnessError(double smoothness_weight, double gridx, double gridy):
    m_smoothness_weight(smoothness_weight),
    m_gridx(gridx), m_gridy(gridy) {}

  template <typename T>
  bool operator()(const T* const bl,   const T* const bottom,    const T* const br,
                  const T* const left, const T* const center,    const T* const right,
                  const T* const tl,   const T* const top,       const T* const tr,
                  T* residuals) const {

    // Normalize by grid size seems to make the functional less
    // sensitive to the actual grid size used.
    residuals[0] = (left[0] + right[0] - 2*center[0])/m_gridx/m_gridx;   // u_xx
    residuals[1] = (br[0] + tl[0] - bl[0] - tr[0] )/4.0/m_gridx/m_gridy; // u_xy
    residuals[2] = residuals[1];                                         // u_yx
    residuals[3] = (bottom[0] + top[0] - 2*center[0])/m_gridy/m_gridy;   // u_yy
    
    for (int i = 0; i < 4; i++)
      residuals[i] *= m_smoothness_weight;

    return true;
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(double smoothness_weight,
                                     double gridx, double gridy){
    return (new ceres::NumericDiffCostFunction<SmoothnessError,
            ceres::CENTRAL, 4, 1, 1, 1, 1, 1, 1, 1, 1, 1>
            (new SmoothnessError(smoothness_weight, gridx, gridy)));
  }

  double m_smoothness_weight, m_gridx, m_gridy;
};

// The gradient error is the sum of squares of
// the first order partial derivatives, with a weight:
// error = gradient_weight * (u_x^2 + u_y^2)

// We will use finite differences to compute these. See
// SmoothnessError() for more details.
struct GradientError {
  GradientError(double gradient_weight, double gridx, double gridy):
    m_gradient_weight(gradient_weight), m_gridx(gridx), m_gridy(gridy) {}

  template <typename T>
  bool operator()(const T* const bottom, const T* const left, const T* const center,
                  const T* const right, const T* const top, T* residuals) const {

    // This results in a smoother solution than using centered differences
    residuals[0] = (right[0]  - center[0])/m_gridx; // u_x
    residuals[1] = (center[0] - left[0]  )/m_gridx; // u_x
    residuals[2] = (top[0]    - center[0])/m_gridy; // u_y
    residuals[3] = (center[0] - bottom[0])/m_gridy; // u_y
    
    for (int i = 0; i < 4; i++)
      residuals[i] *= m_gradient_weight;

    return true;
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(double gradient_weight,
                                     double gridx, double gridy){
    return (new ceres::NumericDiffCostFunction<GradientError,
            ceres::CENTRAL, 4, 1, 1, 1, 1, 1>
            (new GradientError(gradient_weight, gridx, gridy)));
  }

  double m_gradient_weight, m_gridx, m_gridy;
};

// Try to make the DEM in shadow have positive curvature. The error term is
// (curvature_weight *(terrain_xx + terrain_xy - curvature))^2 in the shadow,
// and not used in lit areas.
struct CurvatureInShadowError {
  CurvatureInShadowError(double curvature_in_shadow, double curvature_in_shadow_weight,
                         double gridx, double gridy):
    m_curvature_in_shadow(curvature_in_shadow),
    m_curvature_in_shadow_weight(curvature_in_shadow_weight),
    m_gridx(gridx), m_gridy(gridy) {}

  template <typename T>
  bool operator()(const T* const bottom, const T* const left, const T* const center,
                  const T* const right, const T* const top, T* residuals) const {

    // Normalize by grid size seems to make the functional less
    // sensitive to the actual grid size used.
    double u_xx = (left[0] + right[0] - 2*center[0])/m_gridx/m_gridx;   // u_xx
    double u_yy = (bottom[0] + top[0] - 2*center[0])/m_gridy/m_gridy;   // u_yy
    
    residuals[0] = m_curvature_in_shadow_weight*(u_xx + u_yy - m_curvature_in_shadow);

    return true;
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(double curvature_in_shadow,
                                     double curvature_in_shadow_weight,
                                     double gridx, double gridy){
    return (new ceres::NumericDiffCostFunction<CurvatureInShadowError,
            ceres::CENTRAL, 1, 1, 1, 1, 1, 1>
            (new CurvatureInShadowError(curvature_in_shadow, curvature_in_shadow_weight,
                                        gridx, gridy)));
  }

  double m_curvature_in_shadow, m_curvature_in_shadow_weight, m_gridx, m_gridy;
};

struct SmoothnessErrorPQ {
  SmoothnessErrorPQ(double smoothness_weight_pq, double gridx, double gridy):
    m_smoothness_weight_pq(smoothness_weight_pq),
    m_gridx(gridx), m_gridy(gridy) {}

  template <typename T>
  bool operator()(const T* const bottom_pq, const T* const left_pq, const T* const right_pq,
                  const T* const top_pq, T* residuals) const {

    // Normalize by grid size seems to make the functional less
    // sensitive to the actual grid size used.
    residuals[0] = (right_pq[0] - left_pq[0])/(2*m_gridx);   // p_x
    residuals[1] = (top_pq[0] - bottom_pq[0])/(2*m_gridy);   // p_y
    residuals[2] = (right_pq[1] - left_pq[1])/(2*m_gridx);   // q_x
    residuals[3] = (top_pq[1] - bottom_pq[1])/(2*m_gridy);   // q_y
    
    for (int i = 0; i < 4; i++)
      residuals[i] *= m_smoothness_weight_pq;

    return true;
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(double smoothness_weight_pq,
                                     double gridx, double gridy){
    return (new ceres::NumericDiffCostFunction<SmoothnessErrorPQ,
            ceres::CENTRAL, 4, 2, 2, 2, 2>
            (new SmoothnessErrorPQ(smoothness_weight_pq, gridx, gridy)));
  }

  double m_smoothness_weight_pq, m_gridx, m_gridy;
};

// The integrability error is the discrepancy between the
// independently optimized gradients p and q, and the partial
// derivatives of the dem, here denoted by u.
// error = integrability_weight * ( (u_x - p)^2 + (u_y - q)^2 )

// See SmoothnessError for the notation below. 

struct IntegrabilityError {
  IntegrabilityError(double integrability_weight, double gridx, double gridy):
    m_integrability_weight(integrability_weight),
    m_gridx(gridx), m_gridy(gridy) {}

  template <typename T>
  bool operator()(const T* const bottom, const T* const left, const T* const right,
                  const T* const top, const T* const pq, 
                  T* residuals) const {

    residuals[0] = (right[0] - left[0])/(2*m_gridx) - pq[0];
    residuals[1] = (top[0] - bottom[0])/(2*m_gridy) - pq[1];
    
    for (int i = 0; i < 2; i++)
      residuals[i] *= m_integrability_weight;
    
    return true;
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(double integrability_weight,
                                     double gridx, double gridy){
    return (new ceres::NumericDiffCostFunction<IntegrabilityError,
            ceres::CENTRAL, 2, 1, 1, 1, 1, 2>
            (new IntegrabilityError(integrability_weight, gridx, gridy)));
  }

  double m_integrability_weight, m_gridx, m_gridy;
};


// A cost function that will penalize deviating too much from the original DEM height.
struct HeightChangeError {
  HeightChangeError(double orig_height, double initial_dem_constraint_weight):
    m_orig_height(orig_height), m_initial_dem_constraint_weight(initial_dem_constraint_weight){}

  template <typename T>
  bool operator()(const T* const center, T* residuals) const {
    residuals[0] = (center[0] - m_orig_height)*m_initial_dem_constraint_weight;
    return true;
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(double orig_height,
                                     double initial_dem_constraint_weight){
    return (new ceres::NumericDiffCostFunction<HeightChangeError,
            ceres::CENTRAL, 1, 1>
            (new HeightChangeError(orig_height, initial_dem_constraint_weight)));
  }

  double m_orig_height, m_initial_dem_constraint_weight;
};

// A cost function that will penalize deviating too much from the initial albedo.
struct AlbedoChangeError {
  AlbedoChangeError(double initial_albedo, double albedo_constraint_weight):
    m_initial_albedo(initial_albedo), m_albedo_constraint_weight(albedo_constraint_weight){}

  template <typename T>
  bool operator()(const T* const center, T* residuals) const {
    residuals[0] = (center[0] - m_initial_albedo)*m_albedo_constraint_weight;
    return true;
  }

  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(double initial_albedo,
                                     double albedo_constraint_weight){
    return (new ceres::NumericDiffCostFunction<AlbedoChangeError,
            ceres::CENTRAL, 1, 1>
            (new AlbedoChangeError(initial_albedo, albedo_constraint_weight)));
  }

  double m_initial_albedo, m_albedo_constraint_weight;
};

// Given a DEM, estimate the median grid size in x and in y in meters.
// Given that the DEM heights are in meters as well, having these grid sizes
// will make it possible to handle heights and grids in same units.
void compute_grid_sizes_in_meters(ImageView<double> const& dem,
                                  GeoReference const& geo,
                                  double nodata_val,
                                  double & gridx, double & gridy){

  // Initialize the outputs
  gridx = 0; gridy = 0;

  // Estimate the median height
  std::vector<double> heights;
  for (int col = 0; col < dem.cols(); col++) {
    for (int row = 0; row < dem.rows(); row++) {
      double h = dem(col, row);
      if (h == nodata_val) continue;
      heights.push_back(h);
    }
  }
  double median_height = 0.0;
  int len = heights.size();
  if (len > 0) {
    std::sort(heights.begin(), heights.end());
    median_height = 0.5*(heights[(len-1)/2] + heights[len/2]);
  }

  // Find the grid sizes by estimating the Euclidean distances
  // between points of a DEM at constant height.
  std::vector<double> gridx_vec, gridy_vec;
  for (int col = 0; col < dem.cols()-1; col++) {
    for (int row = 0; row < dem.rows()-1; row++) {

      // The xyz position at the center grid point
      Vector2 lonlat = geo.pixel_to_lonlat(Vector2(col, row));
      Vector3 lonlat3 = Vector3(lonlat(0), lonlat(1), median_height);
      Vector3 base = geo.datum().geodetic_to_cartesian(lonlat3);

      // The xyz position at the right grid point
      lonlat = geo.pixel_to_lonlat(Vector2(col+1, row));
      lonlat3 = Vector3(lonlat(0), lonlat(1), median_height);
      Vector3 right = geo.datum().geodetic_to_cartesian(lonlat3);

      // The xyz position at the bottom grid point
      lonlat = geo.pixel_to_lonlat(Vector2(col, row+1));
      lonlat3 = Vector3(lonlat(0), lonlat(1), median_height);
      Vector3 bottom = geo.datum().geodetic_to_cartesian(lonlat3);

      gridx_vec.push_back(norm_2(right-base));
      gridy_vec.push_back(norm_2(bottom-base));
    }
  }

  // Median grid size
  if (!gridx_vec.empty()) gridx = gridx_vec[gridx_vec.size()/2];
  if (!gridy_vec.empty()) gridy = gridy_vec[gridy_vec.size()/2];
}

void read_sun_positions_from_list(Options const& opt,
                                  std::vector<ModelParams> &model_params) {

  // Initialize the sun position with something (the planet center)
  int num_images = opt.input_images.size();
  model_params.resize(num_images);
  for (int it = 0; it < num_images; it++) {
    model_params[it].sunPosition = Vector3();  
  }
  
  if (opt.sun_positions_list == "") 
    return; // nothing to do
  
  // First read the positions in a map, as they may be out of order
  std::map<std::string, vw::Vector3> sun_positions_map;
  std::ifstream ifs(opt.sun_positions_list.c_str());
  std::string filename;
  double x, y, z;
  while (ifs >> filename >> x >> y >> z)
    sun_positions_map[filename] = Vector3(x, y, z);

  if (sun_positions_map.size() != opt.input_images.size())
    vw_throw(ArgumentErr() << "Expecting to find in file: " << opt.sun_positions_list
             << " as many sun positions as there are images.\n");
  
  // Put the sun positions in model_params.
  for (int it = 0; it < num_images; it++) {
    auto map_it = sun_positions_map.find(opt.input_images[it]);
    if (map_it == sun_positions_map.end()) 
      vw_throw(ArgumentErr() << "Could not read the Sun position from file: "
               << opt.sun_positions_list << " for image: " << opt.input_images[it] << ".\n");

    model_params[it].sunPosition = map_it->second;
  }
}

Vector3 sun_position_from_camera(boost::shared_ptr<CameraModel> camera) {

  // Remove any adjustment to get to the camera proper
  boost::shared_ptr<CameraModel> ucam = unadjusted_model(camera);

  // Try isis
  IsisCameraModel* isis_cam = dynamic_cast<IsisCameraModel*>(ucam.get());
  if (isis_cam != NULL)
    return isis_cam->sun_position();

  // Try csm
  asp::CsmModel* csm_cam = dynamic_cast<asp::CsmModel*>(ucam.get());
  if (csm_cam != NULL)
    return csm_cam->sun_position();

  // No luck. Later there will be a complaint.
  return vw::Vector3();
}

void handle_arguments(int argc, char *argv[], Options& opt) {
  po::options_description general_options("");
  general_options.add_options()
    ("input-dem,i",  po::value(&opt.input_dems_str),
     "The input DEM(s) to refine using SfS. If more than one, their list should be in quotes.")
    ("image-list", po::value(&opt.image_list)->default_value(""),
     "A file containing the list of images, when they are too many to specify on the command line. Use space or newline as separator. See also --camera-list and --mapprojected-data-list.")
    ("camera-list", po::value(&opt.camera_list)->default_value(""),
     "A file containing the list of cameras, when they are too many to specify on the command "
     "line. If the images have embedded camera information, such as for ISIS, this file must "
     "be empty but must be specified if --image-list is specified.")
    ("output-prefix,o", po::value(&opt.out_prefix),
     "Prefix for output filenames.")
    ("max-iterations,n", po::value(&opt.max_iterations)->default_value(10),
     "Set the maximum number of iterations.")
    ("reflectance-type", po::value(&opt.reflectance_type)->default_value(1),
     "Reflectance type (0 = Lambertian, 1 = Lunar-Lambert, 2 = Hapke, 3 = Experimental extension of Lunar-Lambert, 4 = Charon model (a variation of Lunar-Lambert)).")
    ("smoothness-weight", po::value(&opt.smoothness_weight)->default_value(0.04),
     "The weight given to the cost function term which consists of sums of squares of second-order derivatives. A larger value will result in a smoother solution with fewer artifacts. See also --gradient-weight.")
    ("initial-dem-constraint-weight", po::value(&opt.initial_dem_constraint_weight)->default_value(0),
     "A larger value will try harder to keep the SfS-optimized DEM closer to the initial guess DEM. A value between 0.0001 and 0.001 may work, unless your initial DEM is very unreliable.")
    ("bundle-adjust-prefix", po::value(&opt.bundle_adjust_prefix),
     "Use the camera adjustments obtained by previously running bundle_adjust with this output prefix.")
    ("float-albedo",   po::bool_switch(&opt.float_albedo)->default_value(false)->implicit_value(true),
     "Float the albedo for each pixel. Will give incorrect results if only one image is present. The albedo is normalized, its nominal value is 1.")
    ("float-exposure",   po::bool_switch(&opt.float_exposure)->default_value(false)->implicit_value(true),
     "Float the exposure for each image. Will give incorrect results if only one image is present. It usually gives marginal results.")
    ("float-cameras",   po::bool_switch(&opt.float_cameras)->default_value(false)->implicit_value(true),
     "Float the camera pose for each image except the first one. It is suggested that this option be avoided and bundle adjustment be used instead.")
    ("float-all-cameras",   po::bool_switch(&opt.float_all_cameras)->default_value(false)->implicit_value(true),
     "Float the camera pose for each image, including the first one. Experimental. It is suggested to avoid this option.")
    ("model-shadows",   po::bool_switch(&opt.model_shadows)->default_value(false)->implicit_value(true),
     "Model the fact that some points on the DEM are in the shadow (occluded from the Sun).")
    ("compute-exposures-only",   po::bool_switch(&opt.compute_exposures_only)->default_value(false)->implicit_value(true),
     "Quit after saving the exposures. This should be done once for a big DEM, before using these for small sub-clips without recomputing them.")

    ("save-computed-intensity-only",   po::bool_switch(&opt.save_computed_intensity_only)->default_value(false)->implicit_value(true),
     "Save the computed (simulated) image intensities for given DEM, "
     "images, cameras, and reflectance model, without refining the "
     "DEM. The exposures will be computed along the way unless specified "
     "via --image-exposures-prefix, and will be saved to <output prefix>-exposures.txt.")
    
    ("estimate-slope-errors",   po::bool_switch(&opt.estimate_slope_errors)->default_value(false)->implicit_value(true),
     "Estimate the error for each slope (normal to the DEM). This is experimental.")
    ("estimate-height-errors",   po::bool_switch(&opt.estimate_height_errors)->default_value(false)->implicit_value(true),
     "Estimate the SfS DEM height uncertainty by finding the height perturbation (in meters) at each grid point which will make at least one of the simulated images at that point change by more than twice the discrepancy between the unperturbed simulated image and the measured image. The SfS DEM must be provided via the -i option. The number of iterations, blending parameters (--blending-dist, etc.), and smoothness weight are ignored. Results are not computed at image pixels in shadow. This produces <output prefix>-height-error.tif. No SfS DEM is computed.")
    ("height-error-params", po::value(&opt.height_error_params)->default_value(Vector2(5.0,1000.0), "5.0 1000"),
     "Specify the largest height deviation to examine (in meters), and how many samples to use from 0 to that height.")
    ("sun-positions", po::value(&opt.sun_positions_list)->default_value(""),
     "A file having on each line an image name and three values in double precision specifying the Sun position in meters in ECEF coordinates (origin is planet center). Use a space as separator. If not provided, these will be read from the camera files for ISIS and CSM models.")
    ("shadow-thresholds", po::value(&opt.shadow_thresholds)->default_value(""),
     "Optional shadow thresholds for the input images (a list of real values in quotes, one per image).")
    ("shadow-threshold", po::value(&opt.shadow_threshold)->default_value(-1),
     "A shadow threshold to apply to all images instead of using individual thresholds. (Must be positive.)")
    ("custom-shadow-threshold-list", po::value(&opt.custom_shadow_threshold_list)->default_value(""),
     "A list having one image and one shadow threshold per line. For the images specified here, override the shadow threshold supplied by other means with this value.")
    ("max-valid-image-vals", po::value(&opt.max_valid_image_vals)->default_value(""),
     "Optional values for the largest valid image value in each image (a list of real values in quotes, one per image).")
    ("robust-threshold", po::value(&opt.robust_threshold)->default_value(-1.0),
     "If positive, set the threshold for the robust measured-to-simulated intensity difference (using the Cauchy loss). Any difference much larger than this will be penalized. A good value may be 5% to 25% of the average image value or the same fraction of the computed image exposure values.")
    ("albedo-constraint-weight", po::value(&opt.albedo_constraint_weight)->default_value(0),
     "If floating the albedo, a larger value will try harder to keep the optimized albedo close to the nominal value of 1.")
    ("albedo-robust-threshold", po::value(&opt.albedo_robust_threshold)->default_value(0),
     "If floating the albedo and this threshold is positive, apply a Cauchy loss with this threshold to the product of the albedo difference and the albedo constraint weight.")
    ("unreliable-intensity-threshold", po::value(&opt.unreliable_intensity_threshold)->default_value(0.0),
     "Intensities lower than this will be considered unreliable and given less weight.")
    ("skip-images", po::value(&opt.skip_images_str)->default_value(""), "Skip images with these indices (indices start from 0).")
    ("save-dem-with-nodata",   po::bool_switch(&opt.save_dem_with_nodata)->default_value(false)->implicit_value(true),
     "Save a copy of the DEM while using a no-data value at a DEM grid point where all images show shadows. To be used if shadow thresholds are set.")
    ("use-approx-camera-models",   po::bool_switch(&opt.use_approx_camera_models)->default_value(false)->implicit_value(true),
     "Use approximate camera models for speed. Only with ISIS .cub cameras.")
    ("use-rpc-approximation",   po::bool_switch(&opt.use_rpc_approximation)->default_value(false)->implicit_value(true),
     "Use RPC approximations for the camera models instead of approximate tabulated camera models (invoke with --use-approx-camera-models). This is broken and should not be used.")
    ("rpc-penalty-weight", po::value(&opt.rpc_penalty_weight)->default_value(0.1),
     "The RPC penalty weight to use to keep the higher-order RPC coefficients small, if the RPC model approximation is used. Higher penalty weight results in smaller such coefficients.")
    ("rpc-max-error", po::value(&opt.rpc_max_error)->default_value(2),
     "Skip the current camera if the maximum error between a camera model and its RPC approximation is larger than this.")
    ("use-semi-approx",   po::bool_switch(&opt.use_semi_approx)->default_value(false)->implicit_value(true),
     "This is an undocumented experiment.")
    ("coarse-levels", po::value(&opt.coarse_levels)->default_value(0),
     "Solve the problem on a grid coarser than the original by a factor of 2 to this power, then refine the solution on finer grids. It is suggested to not use this option.")
    ("max-coarse-iterations", po::value(&opt.max_coarse_iterations)->default_value(10),
     "How many iterations to do at levels of resolution coarser than the final result.")
    ("crop-input-images",   po::bool_switch(&opt.crop_input_images)->default_value(false)->implicit_value(true),
     "Crop the images to a region that was computed to be large enough, and keep them fully in memory, for speed.")
    ("blending-dist", po::value(&opt.blending_dist)->default_value(0),
     "Give less weight to image pixels close to no-data or boundary values. Enabled only when crop-input-images is true, for performance reasons. Blend over this many pixels.")
    ("blending-power", po::value(&opt.blending_power)->default_value(2.0),
     "A higher value will result in smoother blending.")
    ("min-blend-size", po::value(&opt.min_blend_size)->default_value(0),
     "Do not apply blending in shadowed areas of dimensions less than this.")
    ("allow-borderline-data",   po::bool_switch(&opt.allow_borderline_data)->default_value(false)->implicit_value(true),
     "At the border of the region where there are no lit pixels in any images, do not let the blending weights decay to 0. This noticeably improves the level of detail. The sfs_blend tool may need to be used to further tune this region.")
    ("steepness-factor", po::value(&opt.steepness_factor)->default_value(1.0),
     "Try to make the terrain steeper by this factor. This is not recommended in regular use.")
    ("curvature-in-shadow", po::value(&opt.curvature_in_shadow)->default_value(0.0),
     "Attempt to make the curvature of the DEM (the Laplacian) at points in shadow in all images equal to this value, which should make the DEM curve down.")
    ("curvature-in-shadow-weight", po::value(&opt.curvature_in_shadow_weight)->default_value(0.0),
     "The weight to give to the curvature in shadow constraint.")
    ("lit-curvature-dist", po::value(&opt.lit_curvature_dist)->default_value(0.0),
     "If using a curvature in shadow, start phasing it in this far from the shadow boundary in the lit region (in units of pixels).")
    ("shadow-curvature-dist", po::value(&opt.shadow_curvature_dist)->default_value(0.0),
     "If using a curvature in shadow, have it fully phased in this far from shadow boundary in the shadow region (in units of pixels).")
    ("image-exposures-prefix", po::value(&opt.image_exposure_prefix)->default_value(""),
     "Use this prefix to optionally read initial exposures (filename is <prefix>-exposures.txt).")
    ("model-coeffs-prefix", po::value(&opt.model_coeffs_prefix)->default_value(""),
     "Use this prefix to optionally read model coefficients from a file (filename is <prefix>-model_coeffs.txt).")
    ("model-coeffs", po::value(&opt.model_coeffs)->default_value(""),
     "Use the model coefficients specified as a list of numbers in quotes. Lunar-Lambertian: O, A, B, C, e.g., '1 -0.019 0.000242 -0.00000146'. Hapke: omega, b, c, B0, h, e.g., '0.68 0.17 0.62 0.52 0.52'. Charon: A, f(alpha), e.g., '0.7 0.63'.")
    ("num-haze-coeffs", po::value(&opt.num_haze_coeffs)->default_value(0),
     "Set this to 1 to model the problem as image = exposure * albedo * reflectance + haze, where haze is a single value for each image. This models a small quantity of stray light entering the lens due to scattering and other effects. Use --float-haze to actually optimize the haze (it starts as 0). It will be written as <output-prefix>-haze.txt (ignore all columns of numbers in that file except the first one).")
    ("float-haze",   po::bool_switch(&opt.float_haze)->default_value(false)->implicit_value(true),
     "If specified, float the haze coefficients as part of the optimization, if haze is modeled, so if --num-haze-coeffs is 1.")
    ("haze-prefix", po::value(&opt.image_haze_prefix)->default_value(""),
     "Use this prefix to read initial haze values (filename is <haze-prefix>-haze.txt). The file format is the same as what the tool writes itself, when triggered by the earlier options. If haze is modeled, it will be initially set to 0 unless read from such a file, and will be floated or not depending on whether --float-haze is on. The final haze values will be saved to <output prefix>-haze.txt.")
    ("init-dem-height", po::value(&opt.init_dem_height)->default_value(std::numeric_limits<double>::quiet_NaN()),
     "Use this value for initial DEM heights (measured in meters, relative to the datum). "
     "An input DEM still needs to be provided for georeference information.")
    ("crop-win", po::value(&opt.crop_win)->default_value(BBox2i(0, 0, 0, 0), "xoff yoff xsize ysize"),
     "Crop the input DEM to this region before continuing.")
    ("nodata-value", po::value(&opt.nodata_val)->default_value(std::numeric_limits<double>::quiet_NaN()),
     "Use this as the DEM no-data value, over-riding what is in the initial guess DEM.")
    ("float-dem-at-boundary",   po::bool_switch(&opt.float_dem_at_boundary)->default_value(false)->implicit_value(true),
     "Allow the DEM values at the boundary of the region to also float (not advised).")
    ("boundary-fix",   po::bool_switch(&opt.boundary_fix)->default_value(false)->implicit_value(true),
     "An attempt to let the DEM float at the boundary.")
    ("fix-dem",   po::bool_switch(&opt.fix_dem)->default_value(false)->implicit_value(true),
     "Do not float the DEM at all. Useful when floating the model params.")
    ("float-reflectance-model",   po::bool_switch(&opt.float_reflectance_model)->default_value(false)->implicit_value(true),
     "Allow the coefficients of the reflectance model to float (not recommended).")
    ("float-sun-position",   po::bool_switch(&opt.float_sun_position)->default_value(false)->implicit_value(true),
     "Allow the position of the sun to float.")
    ("integrability-constraint-weight", po::value(&opt.integrability_weight)->default_value(0.0),
     "Use the integrability constraint from Horn 1990 with this value of its weight.")
    ("smoothness-weight-pq", po::value(&opt.smoothness_weight_pq)->default_value(0.00),
     "Smoothness weight for p and q, when the integrability constraint "
     "is used. A larger value will result in a smoother solution "
     "(experimental).")
    ("query",   po::bool_switch(&opt.query)->default_value(false)->implicit_value(true),
     "Print some info and exit. Invoked from parallel_sfs.")
    ("session-type,t",   po::value(&opt.stereo_session)->default_value(""),
     "Select the stereo session type to use for processing. Usually the program can select this automatically by the file extension, except for xml cameras. See the doc for options.")
    ("gradient-weight", po::value(&opt.gradient_weight)->default_value(0.0),
     "The weight given to the cost function term which consists of sums "
     "of squares of first-order derivatives. A larger value will result "
     "in shallower slopes but less noise. This can be used in conjunction with "
     "--smoothness-weight. It is suggested to experiment with this "
     "with a value of 0.0001 - 0.01, while reducing the "
     "smoothness weight to a very small value.")
    ("save-sparingly",   po::bool_switch(&opt.save_sparingly)->default_value(false)->implicit_value(true),
     "Avoid saving any results except the adjustments and the DEM, as that's a lot of files.")
    ("camera-position-step-size", po::value(&opt.camera_position_step_size)->default_value(1.0),
     "Larger step size will result in more aggressiveness in varying the camera position if it is being floated (which may result in a better solution or in divergence).");

  general_options.add( vw::GdalWriteOptionsDescription(opt) );

  po::options_description positional("");
  positional.add_options()
    ("input-images", po::value(&opt.input_images));

  po::positional_options_description positional_desc;
  positional_desc.add("input-images", -1);

  std::string usage("-i <input DEM> -n <max iterations> -o <output prefix> <images> [other options]");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                            positional, positional_desc, usage,
                            allow_unregistered, unregistered);


  if (opt.float_all_cameras)
    opt.float_cameras = true;

  std::istringstream idem(opt.input_dems_str);
  std::string dem;
  while (idem >> dem) opt.input_dems.push_back(dem);
    
  // Sanity checks. Put this early, before separating images from cameras, as that
  // function can print a message not reflecting the true issue of missing the DEM.
  if (opt.input_dems.empty())
    vw_throw( ArgumentErr() << "Missing input DEM(s).\n"
              << usage << general_options );

  // Separate the cameras from the images
  std::vector<std::string> inputs = opt.input_images;

  if (!opt.image_list.empty()) {
    // Read the images and cameras and put them in 'inputs' to be parsed later
    if (opt.camera_list.empty())
      vw_throw(ArgumentErr()
               << "The option --image-list must be invoked together with --camera-list.\n");
    if (!inputs.empty())
      vw_throw(ArgumentErr() << "The option --image-list was specified, but also "
               << "images or cameras on the command line.\n");
    asp::read_list(opt.image_list, inputs);
    std::vector<std::string> tmp;
    asp::read_list(opt.camera_list, tmp);
    for (size_t it = 0; it < tmp.size(); it++) 
      inputs.push_back(tmp[it]);
  }
  
  bool ensure_equal_sizes = true;
  asp::separate_images_from_cameras(inputs,
                                    opt.input_images, opt.input_cameras, // outputs
                                    ensure_equal_sizes); 
  
  if (opt.out_prefix.empty())
    vw_throw( ArgumentErr() << "Missing output prefix.\n"
              << usage << general_options );

  if (opt.max_iterations < 0)
    vw_throw( ArgumentErr() << "The number of iterations must be non-negative.\n"
              << usage << general_options );

  if (opt.input_images.empty())
    vw_throw( ArgumentErr() << "Missing input images.\n"
              << usage << general_options );

  if (opt.smoothness_weight < 0) 
    vw_throw(ArgumentErr() << "Expecting a non-negative smoothness weight.\n");

  if (opt.gradient_weight < 0) 
    vw_throw(ArgumentErr() << "Expecting a non-negative gradient weight.\n");

  if (opt.integrability_weight < 0) 
    vw_throw(ArgumentErr() << "Expecting a non-negative integrability weight.\n");

  if (opt.float_sun_position)
    vw_throw(ArgumentErr() << "Floating sun positions is currently disabled.\n");
  
  if ( opt.float_haze && opt.num_haze_coeffs == 0 ) 
    vw_throw(ArgumentErr() << "Haze cannot be floated unless there is at least one haze coefficient.\n");
  if ( opt.image_haze_prefix != "" && opt.num_haze_coeffs == 0  )
    vw_throw(ArgumentErr() << "Haze cannot be read unless there is at least one haze coefficient.\n");
  
  if (opt.use_rpc_approximation) 
    vw_throw(ArgumentErr() << "The RPC approximation is broken.\n");

  // When we use approximate cameras, and the cameras are fixed, use an approximation
  // for the adjusted camera rather than for the unadjusted one. This uses less memory.
  if (opt.use_approx_camera_models && !opt.float_cameras &&
      !opt.use_rpc_approximation && !opt.use_semi_approx) {
    opt.use_approx_camera_models = false;
    opt.use_approx_adjusted_camera_models = true;
  }

  // Curvature in shadow params
  if (opt.curvature_in_shadow < 0.0 || opt.curvature_in_shadow_weight < 0.0) 
    vw_throw(ArgumentErr() << "Cannot have negative curvature in shadow or its weight.\n");
  if (opt.lit_curvature_dist < 0.0 || opt.shadow_curvature_dist < 0.0) 
    vw_throw(ArgumentErr() << "Cannot have negative curvature distances.\n");    
  if (opt.curvature_in_shadow > 0.0 &&
      opt.shadow_curvature_dist + opt.lit_curvature_dist <= 0.0)
    vw_throw(ArgumentErr() << "When modeling curvature in shadow, expecting a "
             << "positive value of shadow-curvature-dist or list-curvature-dist.\n");    

  if (opt.steepness_factor <= 0.0) 
    vw_throw(ArgumentErr() << "The steepness factor must be positive.\n");    
      
  if (opt.compute_exposures_only){
    if (opt.use_approx_camera_models ||
        opt.use_approx_adjusted_camera_models ||
        opt.use_rpc_approximation ||
        opt.crop_input_images ||
        opt.use_semi_approx) {
      vw_out(WarningMessage) << "When computing exposures only, not using approximate "
                             << "camera models or cropping input images.\n";
      opt.use_approx_camera_models = false;
      opt.use_approx_adjusted_camera_models = false;
      opt.use_rpc_approximation = false;
      opt.crop_input_images = false;
      opt.use_semi_approx = false;
      opt.blending_dist = 0;
      opt.allow_borderline_data = false;
    }

    if (!opt.crop_win.empty()) {
      vw_throw(ArgumentErr() << "When computing exposures only, cannot crop the "
               << "input DEM as that will give wrong results. Use the full DEM.\n");
    }
  }
  
  if (opt.blending_dist > 0 && !opt.crop_input_images) 
    vw_throw(ArgumentErr() << "A blending distance is only supported with --crop-input-images.\n");
  
  if (opt.crop_input_images && (opt.float_cameras || opt.float_all_cameras))  
    vw_throw( ArgumentErr()
              << "Using cropped input images implies that the cameras are not floated.\n" );

  if (opt.allow_borderline_data && !opt.crop_input_images)
    vw_throw(ArgumentErr() << "Option --allow-borderline-data needs option "
             << "--crop-input-images.\n");

  if (opt.allow_borderline_data && opt.blending_dist <= 0) 
    vw::vw_throw(vw::ArgumentErr()
                 << "Option --allow-borderline-data needs a positive --blending-dist.\n");

  if (opt.allow_borderline_data && opt.coarse_levels > 0) 
    vw::vw_throw(vw::ArgumentErr() << "Option --allow-borderline-data cannot be "
                 << "used with multiple coarseness levels.\n");
  
  // Create the output directory
  vw::create_out_dir(opt.out_prefix);

  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);

  // Parse the shadow thresholds
  std::istringstream ist(opt.shadow_thresholds);
  opt.shadow_threshold_vec.clear();
  float val;
  while (ist >> val)
    opt.shadow_threshold_vec.push_back(val);
  if (!opt.shadow_threshold_vec.empty() &&
      opt.shadow_threshold_vec.size() != opt.input_images.size())
    vw_throw(ArgumentErr()
             << "If specified, there must be as many shadow thresholds as images.\n");

  // See if to use opt.shadow_threshold.
  if (opt.shadow_threshold > 0) {
    if (!opt.shadow_threshold_vec.empty())
      vw_throw(ArgumentErr()
               << "Cannot specify both --shadow-threshold and --shadow-thresholds.\n");
    while (opt.shadow_threshold_vec.size() < opt.input_images.size())
      opt.shadow_threshold_vec.push_back(opt.shadow_threshold);
  }

  // Default thresholds are the smallest float.
  // Maybe it should be 0?
  if (opt.shadow_threshold_vec.empty()) {
    for (size_t i = 0; i < opt.input_images.size(); i++) {
      opt.shadow_threshold_vec.push_back(-std::numeric_limits<float>::max());
    }
  }

  // Override some shadow thresholds with custom versions
  if (!opt.custom_shadow_threshold_list.empty()) {
    std::map<std::string, double> custom_thresh;
    std::ifstream ifs(opt.custom_shadow_threshold_list);
    std::string image;
    double val;
    while (ifs >> image >> val) {
      custom_thresh[image] = val;
    }
    
    if (custom_thresh.empty()) 
      vw_throw(ArgumentErr() << "Could not read any data from: "
               << opt.custom_shadow_threshold_list << "\n");
    
    for (size_t it = 0; it < opt.input_images.size(); it++) {
      auto key = custom_thresh.find(opt.input_images[it]);
      if (key != custom_thresh.end()) {
        vw_out() << "Over-riding the shadow threshold for " << opt.input_images[it]
                 << " with: " << key->second << std::endl;
        opt.shadow_threshold_vec[it] = key->second;
      }
    }
  }
  
  // Parse max valid image vals
  std::istringstream ism(opt.max_valid_image_vals);
  opt.max_valid_image_vals_vec.clear();
  while (ism >> val)
    opt.max_valid_image_vals_vec.push_back(val);
  if (!opt.max_valid_image_vals_vec.empty() &&
      opt.max_valid_image_vals_vec.size() != opt.input_images.size())
    vw_throw(ArgumentErr()
             << "If specified, there must be as many max valid image vals as images.\n");

  if (opt.max_valid_image_vals_vec.empty()) {
    for (size_t i = 0; i < opt.input_images.size(); i++) {
      opt.max_valid_image_vals_vec.push_back(std::numeric_limits<float>::max());
    }
  }

  for (size_t i = 0; i < opt.input_images.size(); i++) {
    vw_out() << "Shadow threshold and max valid value for " << opt.input_images[i] << ' '
             << opt.shadow_threshold_vec[i] << ' ' << opt.max_valid_image_vals_vec[i] << std::endl;
  }

  // Initial image exposures, if provided. First read them in a map,
  // as perhaps the initial exposures were created using more images
  // than what we have here. 
  std::string exposure_file = exposure_file_name(opt.image_exposure_prefix);
  opt.image_exposures_vec.clear();
  std::map<std::string, double> img2exp;
  std::string name;
  double dval;
  std::ifstream ise(exposure_file.c_str());
  int exp_count = 0;
  while (ise >> name >> dval){
    img2exp[name] = dval;
    exp_count++;
  }
  ise.close();
  if (exp_count > 0) {
    vw_out() << "Using exposures from: " << exposure_file << std::endl;
    for (size_t i = 0; i < opt.input_images.size(); i++) {
      std::string img = opt.input_images[i];
      std::map<std::string, double>::iterator it = img2exp.find(img);
      if (it == img2exp.end()) {
        vw_throw(ArgumentErr()
                 << "Could not find the exposure for image: " << img << ".\n");
      }
      double exp_val = it->second;
      vw_out() << "Exposure for " << img << ": " << exp_val << std::endl;
      opt.image_exposures_vec.push_back(exp_val);
    }
  }
  if (opt.image_exposure_prefix != "" && exp_count == 0) 
    vw_throw(ArgumentErr()
             << "Could not find the exposures file: " << exposure_file << ".\n");
  
  if (opt.steepness_factor != 1.0) 
    vw_out() << "Making the terrain artificially steeper by factor: " << opt.steepness_factor
             << ".\n";
  
  // Initial image haze, if provided. First read them in a map,
  // as perhaps the initial haze were created using more images
  // than what we have here. 
  if (opt.num_haze_coeffs > 0) {
    std::string haze_file = haze_file_name(opt.image_haze_prefix);
    opt.image_haze_vec.clear();
    std::map< std::string, std::vector<double> > img2haze;
    std::ifstream ish(haze_file.c_str());
    int haze_count = 0;
    while(1){
      std::string line;
      std::getline(ish, line);
      std::istringstream hstream(line);
      if (! (hstream >> name) ) break;
      std::vector<double> haze_vec;
      while (hstream >> dval)
        haze_vec.push_back(dval);
      if (haze_vec.empty()) break;
      haze_count++;

      // Pad the haze vec
      while (haze_vec.size() < g_max_num_haze_coeffs) haze_vec.push_back(0);
      
      img2haze[name] = haze_vec;

      // All haze coefficients beyond the first num_haze_coeffs must
      // be zero, as that means we are reading results written with
      // different number of haze coeffs.
      for (size_t hiter = opt.num_haze_coeffs; hiter < g_max_num_haze_coeffs; hiter++) {
        if (haze_vec[hiter] != 0) 
          vw_throw(ArgumentErr() 
                   << "Found unexpected non-zero haze coefficient: " << haze_vec[hiter] << ".\n");
      }
      
    }
    ish.close();
    if (opt.image_haze_prefix != "" && haze_count == 0) 
      vw_throw(ArgumentErr()
               << "Could not find the haze file: " << haze_file << ".\n");

    
    if (haze_count > 0) {
      vw_out() << "Using haze from: " << haze_file << std::endl;
      for (size_t i = 0; i < opt.input_images.size(); i++) {
        std::string img = opt.input_images[i];
        std::map< std::string, std::vector<double> >::iterator it = img2haze.find(img);
        if (it == img2haze.end()) {
          vw_throw(ArgumentErr()
                   << "Could not find the haze for image: " << img << ".\n");
        }
        std::vector<double> haze_vec = it->second;
        vw_out() << "Haze for " << img << ":";
        for (size_t hiter = 0; hiter < haze_vec.size(); hiter++) 
          vw_out() << " " << haze_vec[hiter];
        vw_out() << std::endl;
        opt.image_haze_vec.push_back(haze_vec);
      }
    }
  }
  
  // Initial model coeffs, if passed on the command line
  if (opt.model_coeffs != "") {
    vw_out() << "Parsing model coefficients: " << opt.model_coeffs << std::endl;
    std::istringstream is(opt.model_coeffs);
    double val;
    while( is >> val){
      opt.model_coeffs_vec.push_back(val);
    }
  }

  // Initial model coefficients, if provided in the file
  if (opt.model_coeffs_prefix != "") {
    std::string model_coeffs_file = model_coeffs_file_name(opt.model_coeffs_prefix);
    vw_out() << "Reading model coefficients from file: " << model_coeffs_file << std::endl;
    std::ifstream ism(model_coeffs_file.c_str());
    opt.model_coeffs_vec.clear();
    while (ism >> dval)
      opt.model_coeffs_vec.push_back(dval);
    ism.close();
    if ( opt.model_coeffs_vec.empty()) {
      vw_throw(ArgumentErr() << "Could not read model coefficients from: " << model_coeffs_file << ".\n");
    }
  }

  if (!opt.model_coeffs_vec.empty()) {
    // Pad with zeros if needed, as the Lunar Lambertian has 4 params, while Hapke has 5 of them.
    // the Charon one has 2.
    while (opt.model_coeffs_vec.size() < g_num_model_coeffs)
      opt.model_coeffs_vec.push_back(0);

    if (opt.model_coeffs_vec.size() != g_num_model_coeffs)
      vw_throw(ArgumentErr()
               << "If specified, there must be " << g_num_model_coeffs << " coefficients.\n");
  }

  // Sanity check
  if (opt.camera_position_step_size <= 0) {
    vw_throw(ArgumentErr() << "Expecting a positive value for camera-position-step-size.\n");
  }

  if (opt.coarse_levels < 0) {
    vw_throw(ArgumentErr() << "Expecting the number of levels to be non-negative.\n");
  }

  // Need this to be able to load adjusted camera models. That will happen
  // in the stereo session.
  asp::stereo_settings().bundle_adjust_prefix = opt.bundle_adjust_prefix;

  if (opt.input_images.size() <= 1 && opt.float_albedo && 
      opt.initial_dem_constraint_weight <= 0 && opt.albedo_constraint_weight <= 0.0)
    vw_throw(ArgumentErr()
             << "Floating the albedo is ill-posed for just one image without "
             << "the initial DEM constraint or the albedo constraint.\n");

  if (opt.input_images.size() <= 1 && opt.float_exposure && opt.initial_dem_constraint_weight <= 0)
    vw_throw(ArgumentErr()
             << "Floating the exposure is ill-posed for just one image.\n");

  if (opt.input_images.size() <= 1 && opt.float_dem_at_boundary)
    vw_throw(ArgumentErr()
             << "Floating the DEM at the boundary is ill-posed for just one image.\n");

  if (opt.boundary_fix && opt.integrability_weight == 0) {
    vw_throw(ArgumentErr()
             << "The boundary fix only works with the integrability constraint.\n");
  }
  
  // Start with given images to skip. Later, for each dem clip, we may
  // skip more, if those images do not overlap with the clip.
  int num_dems = opt.input_dems.size();
  opt.skip_images.resize(num_dems);
  std::set<int> curr_skip_images;
  if (opt.skip_images_str != "") {
    std::istringstream is(opt.skip_images_str);
    int val;
    while( is >> val)
      curr_skip_images.insert(val);
  }
  for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
    opt.skip_images[dem_iter] = curr_skip_images;
  }

  if (opt.estimate_slope_errors && opt.estimate_height_errors) 
    vw_throw( ArgumentErr() << "Cannot estimate both slope and height error at the same time.");

  if (opt.estimate_height_errors && opt.model_shadows) 
    vw_throw( ArgumentErr() << "Cannot estimate height error when modeling shadows.");
  
  if (opt.save_computed_intensity_only || opt.estimate_slope_errors || opt.estimate_height_errors){
    if (opt.max_iterations > 0 || opt.max_coarse_iterations > 0){
      vw_out(WarningMessage) << "Using 0 iterations.\n";
      opt.max_iterations = 0;
      opt.max_coarse_iterations = 0;
    }
    //if (!opt.model_shadows) 
    //  vw_out(WarningMessage) << "It is suggested that --model-shadows be used.\n";

    if (opt.coarse_levels > 0) {
      vw_out(WarningMessage) << "Using 0 coarse levels.\n";
      opt.coarse_levels = 0;
    }

    if (opt.use_approx_camera_models ||
        opt.use_approx_adjusted_camera_models ||
        opt.use_rpc_approximation ||
        opt.crop_input_images) {
      vw_out(WarningMessage) << "Not using approximate camera models or cropping input images.\n";
      opt.use_approx_camera_models = false;
      opt.use_approx_adjusted_camera_models = false;
      opt.use_rpc_approximation = false;
      opt.crop_input_images = false;
      opt.use_semi_approx = false;
    }
    
    //if (opt.image_exposures_vec.empty())
    //  vw_throw( ArgumentErr()
    // << "Expecting the exposures to be computed and passed in.\n" );
    
    if (opt.num_haze_coeffs > 0 && opt.image_haze_vec.empty())
      vw_throw( ArgumentErr()
                << "Expecting the haze to be computed and passed in.\n" );
  }
  
  
}

// Run sfs at a given coarseness level
void run_sfs_level(// Fixed inputs
                   int num_iterations, Options & opt,
                   std::vector<GeoReference> const& geo,
                   double smoothness_weight,
                   double dem_nodata_val,
                   std::vector< std::vector<BBox2i>>     const& crop_boxes,
                   std::vector< std::vector<MaskedImgT>> const& masked_images,
                   std::vector< std::vector<DoubleImgT>> const& blend_weights,
                   GlobalParams const& global_params,
                   std::vector<ModelParams> const & model_params,
                   std::vector< ImageView<double>> const& orig_dems, 
                   double initial_albedo,
                   ImageView<int> const& lit_image_mask,
                   ImageView<double> const& curvature_in_shadow_weight,
                   // Quantities that will float
                   std::vector< ImageView<double>> & dems,
                   std::vector< ImageView<double>> & albedos,
                   std::vector< std::vector<boost::shared_ptr<CameraModel>>> & cameras,
                   std::vector<double> & exposures,
                   std::vector< std::vector<double>> & haze,
                   std::vector<double> & scaled_sun_posns,
                   std::vector<double> & adjustments,
                   std::vector<double> & reflectance_model_coeffs){

  int num_images = opt.input_images.size();
  int num_dems   = dems.size();
  ceres::Problem problem;
  
  // Find the grid sizes in meters. Note that dem heights are in
  // meters too, so we treat both horizontal and vertical measurements
  // in same units.
  double gridx, gridy;
  compute_grid_sizes_in_meters(dems[0], geo[0], dem_nodata_val, gridx, gridy);
  vw_out() << "grid in x and y in meters: "
           << gridx << ' ' << gridy << std::endl;
  g_gridx = &gridx;
  g_gridy = &gridy;

  std::vector<double> max_dem_height(num_dems, -std::numeric_limits<double>::max());
  if (opt.model_shadows) {
    // Find the max DEM height
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) { 
      double curr_max_dem_height = -std::numeric_limits<double>::max();
      for (int col = 0; col < dems[dem_iter].cols(); col++) {
        for (int row = 0; row < dems[dem_iter].rows(); row++) {
          if (dems[dem_iter](col, row) > curr_max_dem_height) {
            curr_max_dem_height = dems[dem_iter](col, row);
          }
        }
      }
      max_dem_height[dem_iter] = curr_max_dem_height;
    }
  }
  g_max_dem_height = &max_dem_height;

  // See if a given image is used in at least one clip or skipped in
  // all of them
  std::vector<bool> use_image(num_images, false);
  int num_used = 0;
  for (int image_iter = 0; image_iter < num_images; image_iter++) {
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      if (opt.skip_images[dem_iter].find(image_iter) == opt.skip_images[dem_iter].end()){
        use_image[image_iter] = true;
        num_used++;
      }
    }
  }

  // We define p and q as the partial derivatives in x in y of the dem.
  // When using the integrability constraint, they are floated as variables
  // in their own right, while constrained to not go too far from the DEM.
  std::vector< ImageView<Vector2> > pq;  
  pq.resize(num_dems);
  if (opt.integrability_weight > 0) {
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      pq[dem_iter].set_size(dems[dem_iter].cols(), dems[dem_iter].rows());

      for (int col = 1; col < dems[dem_iter].cols()-1; col++) {
        for (int row = 1; row < dems[dem_iter].rows()-1; row++) {
          // Note that the top value is dems[dem_iter](col, row-1) and the 
          //            bottom value is dems[dem_iter](col, row+1).
          pq[dem_iter](col, row)[0] // same as (right - left)/(2*gridx)
            = (dems[dem_iter](col+1, row) - dems[dem_iter](col-1, row))/(2*gridx);
          pq[dem_iter](col, row)[1] // same as (top - bottom)/(2*gridy)
            = (dems[dem_iter](col, row-1) - dems[dem_iter](col, row+1))/(2*gridy);
        }
      }
    }
  }

  // Use a simpler cost function if only the DEM is floated. Not sure how much
  // of speedup that gives.
  bool float_dem_only = true;
  if (opt.float_albedo || opt.float_exposure || opt.float_cameras ||
      opt.float_all_cameras || opt.float_dem_at_boundary ||
      opt.boundary_fix || opt.fix_dem || opt.float_reflectance_model ||
      opt.float_sun_position || opt.float_haze || opt.integrability_weight > 0){
    float_dem_only = false;
  }
  
  std::set<int> use_dem, use_albedo; // to avoid a crash in Ceres when a param is fixed but not set
  
  for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
    
    int bd = 1;
    if (opt.boundary_fix) bd = 0;
    
    // Add a residual block for every grid point not at the boundary
    for (int col = bd; col < dems[dem_iter].cols()-bd; col++) {
      for (int row = bd; row < dems[dem_iter].rows()-bd; row++) {
        
        // Intensity error for each image
        for (int image_iter = 0; image_iter < num_images; image_iter++) {

          if (opt.skip_images[dem_iter].find(image_iter) != opt.skip_images[dem_iter].end()) {
            continue;
          }
          
          ceres::LossFunction* loss_function_img = NULL;
          if (opt.robust_threshold > 0) 
            loss_function_img = new ceres::CauchyLoss(opt.robust_threshold);
          
          if (float_dem_only) {
            ceres::CostFunction* cost_function_img =
              IntensityErrorFloatDemOnly::Create(col, row,
                                                 dems[dem_iter],
                                                 albedos[dem_iter](col, row), 
                                                 &reflectance_model_coeffs[0],
                                                 &exposures[image_iter],      // exposure
                                                 &haze[image_iter][0],        // haze
                                                 &adjustments[6*image_iter],  // camera adjustments
                                                 geo[dem_iter],
                                                 opt.model_shadows,
                                                 opt.camera_position_step_size,
                                                 max_dem_height[dem_iter],
                                                 gridx, gridy,
                                                 global_params, model_params[image_iter],
                                                 crop_boxes[dem_iter][image_iter],
                                                 masked_images[dem_iter][image_iter],
                                                 blend_weights[dem_iter][image_iter],
                                                 &scaled_sun_posns[3*image_iter], // sun positions
                                                 cameras[dem_iter][image_iter]);
            problem.AddResidualBlock(cost_function_img, loss_function_img,
                                     &dems[dem_iter](col-1, row),  // left
                                     &dems[dem_iter](col, row),    // center
                                     &dems[dem_iter](col+1, row),  // right
                                     &dems[dem_iter](col, row+1),  // bottom
                                     &dems[dem_iter](col, row-1)  // top
                                     );
            use_dem.insert(dem_iter); 
            
          }else if (opt.integrability_weight == 0){
            ceres::CostFunction* cost_function_img =
              IntensityError::Create(col, row, dems[dem_iter], geo[dem_iter],
                                     opt.model_shadows,
                                     opt.camera_position_step_size,
                                     max_dem_height[dem_iter],
                                     gridx, gridy,
                                     global_params, model_params[image_iter],
                                     crop_boxes[dem_iter][image_iter],
                                     masked_images[dem_iter][image_iter],
                                     blend_weights[dem_iter][image_iter],
                                     &scaled_sun_posns[3*image_iter], // sun positions
                                     cameras[dem_iter][image_iter]);
            problem.AddResidualBlock(cost_function_img, loss_function_img,
                                     &exposures[image_iter],       // exposure
                                     &haze[image_iter][0],         // haze
                                     &dems[dem_iter](col-1, row),  // left
                                     &dems[dem_iter](col, row),    // center
                                     &dems[dem_iter](col+1, row),  // right
                                     &dems[dem_iter](col, row+1),  // bottom
                                     &dems[dem_iter](col, row-1),  // top
                                     &albedos[dem_iter](col, row), // albedo
                                     &adjustments[6*image_iter],   // camera
                                     //&scaled_sun_posns[3*image_iter], // sun positions
                                     &reflectance_model_coeffs[0]);
            use_dem.insert(dem_iter); 
            use_albedo.insert(dem_iter);
          } else {
            // Use the integrability constraint
            ceres::CostFunction* cost_function_img =
              IntensityErrorPQ::Create(col, row, dems[dem_iter], geo[dem_iter],
                                       opt.model_shadows,
                                       opt.camera_position_step_size,
                                       max_dem_height[dem_iter],
                                       gridx, gridy,
                                       global_params, model_params[image_iter],
                                       crop_boxes[dem_iter][image_iter],
                                       masked_images[dem_iter][image_iter],
                                       blend_weights[dem_iter][image_iter],
                                       cameras[dem_iter][image_iter]);
            problem.AddResidualBlock(cost_function_img, loss_function_img,
                                     &exposures[image_iter],          // exposure
                                     &haze[image_iter][0],            // haze
                                     &dems[dem_iter](col, row),       // center
                                     &pq[dem_iter](col, row)[0],      // pq
                                     &albedos[dem_iter](col, row),    // albedo
                                     &adjustments[6*image_iter],      // camera
                                     &scaled_sun_posns[3*image_iter], // sun positions
                                     &reflectance_model_coeffs[0]);   // reflectance 
            
            
            use_dem.insert(dem_iter); 
            use_albedo.insert(dem_iter);
          }
          
        } // end iterating over images
        
        if (col > 0 && col < dems[dem_iter].cols()-1 &&
            row > 0 && row < dems[dem_iter].rows()-1 ) {
          
          // Smoothness penalty. We always add this, even if the weight is 0,
          // to make Ceres not complain about blocks not being set. 
          ceres::LossFunction* loss_function_sm = NULL;
          ceres::CostFunction* cost_function_sm =
            SmoothnessError::Create(smoothness_weight, gridx, gridy);
          problem.AddResidualBlock(cost_function_sm, loss_function_sm,
                                   &dems[dem_iter](col-1, row+1),  // bottom left
                                   &dems[dem_iter](col, row+1),    // bottom 
                                   &dems[dem_iter](col+1, row+1),  // bottom right
                                   &dems[dem_iter](col-1, row  ),  // left
                                   &dems[dem_iter](col, row  ),    // center
                                   &dems[dem_iter](col+1, row  ),  // right 
                                   &dems[dem_iter](col-1, row-1),  // top left
                                   &dems[dem_iter](col, row-1),    // top
                                   &dems[dem_iter](col+1, row-1)); // top right

          // Add curvature in shadow. Note that we use a per-pixel curvature_in_shadow_weight,
          // to gradually phase it in to avoid artifacts.
          if (opt.curvature_in_shadow_weight > 0.0 && curvature_in_shadow_weight(col, row) > 0) {
            ceres::LossFunction* loss_function_cv = NULL;
            ceres::CostFunction* cost_function_cv =
              CurvatureInShadowError::Create(opt.curvature_in_shadow,
                                             curvature_in_shadow_weight(col, row),
                                             gridx, gridy);
            problem.AddResidualBlock(cost_function_cv, loss_function_cv,
                                     &dems[dem_iter](col,   row+1),  // bottom 
                                     &dems[dem_iter](col-1, row),    // left
                                     &dems[dem_iter](col,   row),    // center
                                     &dems[dem_iter](col+1, row),    // right 
                                     &dems[dem_iter](col,   row-1)); // top
          }

          // Add gradient weight
          if (opt.gradient_weight > 0.0) {
            ceres::LossFunction* loss_function_grad = NULL;
            ceres::CostFunction* cost_function_grad =
              GradientError::Create(opt.gradient_weight, gridx, gridy);
            problem.AddResidualBlock(cost_function_grad, loss_function_grad,
                                     &dems[dem_iter](col,   row+1),  // bottom 
                                     &dems[dem_iter](col-1, row),    // left
                                     &dems[dem_iter](col,   row),    // center
                                     &dems[dem_iter](col+1, row),    // right 
                                     &dems[dem_iter](col,   row-1)); // top
          }
        
          if (opt.integrability_weight > 0) {
            ceres::LossFunction* loss_function_int = NULL;
            ceres::CostFunction* cost_function_int =
              IntegrabilityError::Create(opt.integrability_weight, gridx, gridy);
            problem.AddResidualBlock(cost_function_int, loss_function_int,
                                     &dems[dem_iter](col,   row+1),   // bottom
                                     &dems[dem_iter](col-1, row),     // left
                                     &dems[dem_iter](col+1, row),     // right
                                     &dems[dem_iter](col,   row-1),   // top
                                     &pq[dem_iter]  (col,   row)[0]); // pq

            if (opt.smoothness_weight_pq > 0) {
              ceres::LossFunction* loss_function_sm_pq = NULL;
              ceres::CostFunction* cost_function_sm_pq =
                SmoothnessErrorPQ::Create(opt.smoothness_weight_pq, gridx, gridy);
              problem.AddResidualBlock(cost_function_sm_pq, loss_function_sm_pq,
                                       &pq[dem_iter](col, row+1)[0],  // bottom 
                                       &pq[dem_iter](col-1, row)[0],  // left
                                       &pq[dem_iter](col+1, row)[0],  // right 
                                       &pq[dem_iter](col, row-1)[0]); // top
            }
          }
          
          use_dem.insert(dem_iter); 
          
          // Deviation from prescribed height constraint
          if (opt.initial_dem_constraint_weight > 0) {
            ceres::LossFunction* loss_function_hc = NULL;
            ceres::CostFunction* cost_function_hc =
              HeightChangeError::Create(orig_dems[dem_iter](col, row),
                                        opt.initial_dem_constraint_weight);
            problem.AddResidualBlock(cost_function_hc, loss_function_hc,
                                     &dems[dem_iter](col, row));
            use_dem.insert(dem_iter); 
          }
          
          // Deviation from prescribed albedo
          if (opt.float_albedo > 0 && opt.albedo_constraint_weight > 0) {
            
            ceres::LossFunction* loss_function_hc = NULL;
            if (opt.albedo_robust_threshold > 0)
              loss_function_hc = new ceres::CauchyLoss(opt.albedo_robust_threshold);
            ceres::CostFunction* cost_function_hc =
              AlbedoChangeError::Create(initial_albedo,
                                        opt.albedo_constraint_weight);
            problem.AddResidualBlock(cost_function_hc, loss_function_hc,
                                     &albedos[dem_iter](col, row));
            use_albedo.insert(dem_iter);
          }
        }
        
      } // end row iter
    } // end col iter
    
    // DEM at the boundary must be fixed.
    if (!opt.float_dem_at_boundary) {
      for (int col = 0; col < dems[dem_iter].cols(); col++) {
        for (int row = 0; row < dems[dem_iter].rows(); row++) {
          if (col == 0 || col == dems[dem_iter].cols() - 1 ||
              row == 0 || row == dems[dem_iter].rows() - 1 ) {
            if (use_dem.find(dem_iter) != use_dem.end())
              problem.SetParameterBlockConstant(&dems[dem_iter](col, row));
          }
        }
      }
    }
    
    if (opt.fix_dem) {
      for (int col = 0; col < dems[dem_iter].cols(); col++) {
        for (int row = 0; row < dems[dem_iter].rows(); row++) {
          if (use_dem.find(dem_iter) != use_dem.end())
            problem.SetParameterBlockConstant(&dems[dem_iter](col, row));
        }
      }
    }
    
    if (opt.initial_dem_constraint_weight <= 0 && num_used <= 1) {    

      if (opt.float_albedo && opt.albedo_constraint_weight <= 0) {
        vw_out() << "No DEM or albedo constraint is used, and there is at most one "
                 << "usable image. Fixing the albedo.\n";
        opt.float_albedo = false;
      }

      // If there's just one image, don't float the exposure, as the
      // problem is under-determined. If we float the albedo, we will
      // implicitly float the exposure, hence keep the exposure itself
      // fixed.
      if (opt.float_exposure) {
        vw_out() << "No DEM constraint is used, and there is at most one "
                 << "usable image. Fixing the exposure.\n";
        opt.float_exposure = false;
      }
    }
    
    // If to float the albedo
    if (!float_dem_only) {
      for (int col = 1; col < dems[dem_iter].cols() - 1; col++) {
        for (int row = 1; row < dems[dem_iter].rows() - 1; row++) {
          if (!opt.float_albedo && num_used > 0 && use_albedo.find(dem_iter) != use_albedo.end())
            problem.SetParameterBlockConstant(&albedos[dem_iter](col, row));
        }
      }
    }
    
  } // end iterating over DEMs

  if (!float_dem_only) {

    // If floating the DEM only, none of the below parameters are even added to the problem,
    // it does not make sense to check to keep them fixed or floating them.
    
    if (!opt.float_exposure){
      for (int image_iter = 0; image_iter < num_images; image_iter++) {
        if (use_image[image_iter]) problem.SetParameterBlockConstant(&exposures[image_iter]);
      }
    }
    if (!opt.float_haze){
      for (int image_iter = 0; image_iter < num_images; image_iter++) {
        if (use_image[image_iter]) problem.SetParameterBlockConstant(&haze[image_iter][0]);
      }
    }
    
    if (!opt.float_cameras) {
      vw_out() << "Not floating cameras." << std::endl;
      for (int image_iter = 0; image_iter < num_images; image_iter++){
        if (use_image[image_iter]){
          problem.SetParameterBlockConstant(&adjustments[6*image_iter]);
        }
      }
    }else if (!opt.float_all_cameras){
      // Fix the first camera, let the other ones conform to it.
      // TODO: This needs further study.
      vw_out() << "Floating all cameras sans the first one." << std::endl;
      int image_iter = 0;
      if (use_image[image_iter]){
        problem.SetParameterBlockConstant(&adjustments[6*image_iter]);
      }
      
    }else{
      vw_out() << "Floating all cameras, including the first one." << std::endl;
    }

    // If to float the reflectance model coefficients
    if (!opt.float_reflectance_model && num_used > 0) {
      problem.SetParameterBlockConstant(&reflectance_model_coeffs[0]);
    }
    
    if (!opt.float_sun_position){
      if (opt.integrability_weight != 0){
        for (int image_iter = 0; image_iter < num_images; image_iter++) {
          if (use_image[image_iter]) problem.SetParameterBlockConstant(&scaled_sun_posns[3*image_iter]);
        }
      }
    }
  }
  
  if (opt.num_threads > 1 &&
      opt.stereo_session == "isis"  &&
      !opt.use_approx_camera_models &&
      !opt.use_approx_adjusted_camera_models) {
    vw_out() << "Using exact ISIS camera models. Can run with only a single thread.\n";
    opt.num_threads = 1;
  }

  vw_out() << "Using: " << opt.num_threads << " thread(s).\n";

  ceres::Solver::Options options;
  options.gradient_tolerance = 1e-16;
  options.function_tolerance = 1e-16;
  options.max_num_iterations = num_iterations;
  options.minimizer_progress_to_stdout = 1;
  options.num_threads = opt.num_threads;
  options.linear_solver_type = ceres::SPARSE_SCHUR;

  // Use a callback function at every iteration
  SfsCallback callback;
  options.callbacks.push_back(&callback);
  options.update_state_every_iteration = true;

  // A bunch of global variables to use in the callback
  g_dem            = &dems;
  g_pq             = &pq;
  g_albedo         = &albedos;
  g_geo            = &geo;
  g_global_params  = &global_params;
  g_model_params   = &model_params;
  g_crop_boxes     = &crop_boxes;
  g_masked_images  = &masked_images;
  g_blend_weights  = &blend_weights;
  g_cameras        = &cameras;
  g_iter           = -1; // reset the iterations for each level
  g_final_iter     = false;

  // Solve the problem if asked to do iterations. Otherwise
  // just keep the DEM at the initial guess, while saving
  // all the output data as if iterations happened.
  ceres::Solver::Summary summary;
  if (options.max_num_iterations > 0)
    ceres::Solve(options, &problem, &summary);

  // Save the final results
  g_final_iter = true;
  ceres::IterationSummary callback_summary;
  callback(callback_summary);
  
  vw_out() << summary.FullReport() << "\n" << std::endl;

  // callTop();
}

#if 0

// Function for highlighting no-data
template<class PixelT>
class NoData {
  typedef typename CompoundChannelType<PixelT>::type channel_type;
  channel_type m_nodata;
  typedef ChannelRange<channel_type> range_type;
public:
  NoData( channel_type nodata ) : m_nodata(nodata) {}

  template <class Args> struct result {
    typedef channel_type type;
  };

  inline channel_type operator()( channel_type const& val ) const {
    return (!(val != m_nodata && !std::isnan(val)))? range_type::max() : range_type::min();
  }
};

template <class ImageT, class NoDataT>
UnaryPerPixelView<ImageT,UnaryCompoundFunctor<NoData<typename ImageT::pixel_type>, typename ImageT::pixel_type>  >
inline nodata( ImageViewBase<ImageT> const& image, NoDataT nodata ) {
  typedef UnaryCompoundFunctor<NoData<typename ImageT::pixel_type>, typename ImageT::pixel_type> func_type;
  func_type func( nodata );
  return UnaryPerPixelView<ImageT,func_type>( image.impl(), func );
}

#endif

#if 0
// Prototype code to identify permanently shadowed areas
// and deepen the craters there. Needs to be integrated
// and tested with various shapes of the deepened crater.
void deepenCraters() {
  std::vector<std::string> image;

  std::string dem_file = argv[argc - 1];
  
  float dem_nodata_val = -std::numeric_limits<float>::max();
  if (vw::read_nodata_val(dem_file, dem_nodata_val)){
    vw_out() << "Dem nodata: " << dem_nodata_val << std::endl;
  }

  ImageView< PixelMask<float> > dem (create_mask(DiskImageView<float>(dem_file), dem_nodata_val));

  vw::cartography::GeoReference georef;
  if (!read_georeference(georef, dem_file))
    vw_throw( ArgumentErr() << "The input DEM " << dem_file << " has no georeference.\n" );
  
  // The maximum of all valid pixel values with no-data where there is no-valid data.
  ImageView< PixelMask<float> > max_img(dem.cols(), dem.rows());
  for (int col = 0; col < dem.cols(); col++) {
    for (int row = 0; row < dem.rows(); row++) {
      max_img(col, row) = dem_nodata_val;
      max_img(col, row).invalidate();
    }
  }
  
  for (int i = 1; i < argc - 1; i++) {

    std::string img_file = argv[i];
    float img_nodata_val = -std::numeric_limits<float>::max();
    if (vw::read_nodata_val(img_file, img_nodata_val)){
      vw_out() << "Img nodata: " << img_nodata_val << std::endl;
    }
    
    ImageView< PixelMask<float> > img(create_mask(DiskImageView<float>(img_file), img_nodata_val));
    std::cout << "cols and rows are " << img.cols() << ' ' << img.rows() << std::endl;
    if (img.cols() != dem.cols() || img.rows() != dem.rows()) {
      vw_throw(ArgumentErr() << "Images and DEM must have same size.\n");
    }

    for (int col = 0; col < img.cols(); col++) {
      for (int row = 0; row < img.rows(); row++) {

        // Nothing to do if the current image has invalid data
        if (!is_valid(img(col, row)))
          continue; 

        // If the output image is not valid yet, copy the current image's valid pixel
        if (!is_valid(max_img(col, row) && img(col, row).child() > 0)) {
          max_img(col, row) = img(col, row);
          continue;
        }

        // Now both the current image and the output image are valid
        if (img(col, row).child() > max_img(col, row).child() &&
            img(col, row).child() > 0) {
          max_img(col, row) = img(col, row);
        }
        
      }
    }
  }

  // At the boundary the intensity is always invalid, but that is due to
  // computational limitations. Make it valid if we can.
  // TODO: Test here that the image has at least 3 rows and 3 cols!
  for (int col = 0; col < max_img.cols(); col++) {
    for (int row = 0; row < max_img.rows(); row++) {
      if ( (col == 0 || col == max_img.cols() - 1) ||
           (row == 0 || row == max_img.rows() - 1) ) {
        int next_col = col, next_row = row;
        if (col == 0) next_col = 1;
        if (col == max_img.cols() - 1) next_col = max_img.cols() - 2;
        if (row == 0) next_row = 1;
        if (row == max_img.rows() - 1) next_row = max_img.rows() - 2;

        if (!is_valid(max_img(col, row)) && is_valid(max_img(next_col, next_row))) 
          max_img(col, row) = max_img(next_col, next_row);
      }
    }
  }
  
  std::string max_img_file = "max_img.tif";
  bool has_nodata = true, has_georef = true;
  TerminalProgressCallback tpc("", "\t--> ");

  vw_out() << "Writing: " << max_img_file << "\n";
  
  block_write_gdal_image(max_img_file, apply_mask(max_img, dem_nodata_val),
                         has_georef, georef,
                         has_nodata, dem_nodata_val,
                         opt, tpc);

  ImageView<double> grass = grassfire(nodata(select_channel(max_img, 0), dem_nodata_val));

  // Scale as craters are shallow.
  // TODO: Need to think of a better algorithm!
  for (int col = 0; col < grass.cols(); col++) {
    for (int row = 0; row < grass.rows(); row++) {
      grass(col, row) *= 0.2;
    }
  }

  // Blur with a given sigma
  double sigma = atof(getenv("SIGMA"));
  //blur_weights(grass, sigma);
  ImageView<double> blurred_grass;
  if (sigma > 0) 
    blurred_grass = gaussian_filter(grass, sigma);
  else
    blurred_grass = copy(grass);
  
  std::string grass_file = "grass.tif";
  vw_out() << "Writing: " << grass_file << "\n";

  bool grass_has_nodata = false;
  block_write_gdal_image(grass_file, blurred_grass,
                         has_georef, georef,
                         grass_has_nodata, dem_nodata_val,
                         opt, tpc);

  // Bias the DEM by that grassfire height deepening the craters
  for (int col = 0; col < dem.cols(); col++) {
    for (int row = 0; row < dem.rows(); row++) {
      if (is_valid(dem(col, row))) {
        dem(col, row).child() -= blurred_grass(col, row);
      }
    }
  }


  std::string out_dem_file = "out_dem.tif";
  vw_out() << "Writing: " << out_dem_file << "\n";
  
  block_write_gdal_image(out_dem_file, apply_mask(dem, dem_nodata_val),
                         has_georef, georef,
                         has_nodata, dem_nodata_val,
                         opt, tpc);

}
#endif

void setUpModelParams(GlobalParams & global_params, Options & opt) {
  if (opt.reflectance_type == 0)
    global_params.reflectanceType = LAMBERT;
  else if (opt.reflectance_type == 1)
    global_params.reflectanceType = LUNAR_LAMBERT;
  else if (opt.reflectance_type == 2)
    global_params.reflectanceType = HAPKE;
  else if (opt.reflectance_type == 3)
    global_params.reflectanceType = ARBITRARY_MODEL;
  else if (opt.reflectance_type == 4)
    global_params.reflectanceType = CHARON;
  else
    vw_throw( ArgumentErr() << "Expecting Lambertian or Lunar-Lambertian reflectance." );
  global_params.phaseCoeffC1 = 0; 
  global_params.phaseCoeffC2 = 0;
  
  // Default model coefficients, unless they were read already
  if (opt.model_coeffs_vec.empty()) {
    opt.model_coeffs_vec.resize(g_num_model_coeffs);
    if (global_params.reflectanceType == LUNAR_LAMBERT ||
        global_params.reflectanceType == ARBITRARY_MODEL ) {
      // Lunar lambertian or its crazy experimental generalization
      opt.model_coeffs_vec.resize(g_num_model_coeffs);
      opt.model_coeffs_vec[0] = 1;
      opt.model_coeffs_vec[1] = -0.019;
      opt.model_coeffs_vec[2] =  0.000242;   //0.242*1e-3;
      opt.model_coeffs_vec[3] = -0.00000146; //-1.46*1e-6;
      opt.model_coeffs_vec[4] = 1;
      opt.model_coeffs_vec[5] = 0;
      opt.model_coeffs_vec[6] = 0;
      opt.model_coeffs_vec[7] = 0;
      opt.model_coeffs_vec[8] = 1;
      opt.model_coeffs_vec[9] = -0.019;
      opt.model_coeffs_vec[10] =  0.000242;   //0.242*1e-3;
      opt.model_coeffs_vec[11] = -0.00000146; //-1.46*1e-6;
      opt.model_coeffs_vec[12] = 1;
      opt.model_coeffs_vec[13] = 0;
      opt.model_coeffs_vec[14] = 0;
      opt.model_coeffs_vec[15] = 0;
    }else if (global_params.reflectanceType == HAPKE) {
      opt.model_coeffs_vec[0] = 0.68; // omega (also known as w)
      opt.model_coeffs_vec[1] = 0.17; // b
      opt.model_coeffs_vec[2] = 0.62; // c
      opt.model_coeffs_vec[3] = 0.52; // B0
      opt.model_coeffs_vec[4] = 0.52; // h
    }else if (global_params.reflectanceType == CHARON) {
      opt.model_coeffs_vec.resize(g_num_model_coeffs);
      opt.model_coeffs_vec[0] = 0.7; // A
      opt.model_coeffs_vec[1] = 0.63; // f(alpha)
    }else if (global_params.reflectanceType != LAMBERT) {
      vw_throw( ArgumentErr() << "The Hapke model coefficients were not set. "
                << "Use the --model-coeffs option." );
    }
  }
}

int main(int argc, char* argv[]) {
  
  Stopwatch sw_total;
  sw_total.start();
  
  Options opt;
  g_opt = &opt;
  try {
    handle_arguments(argc, argv, opt);

    if (opt.compute_exposures_only && !opt.image_exposures_vec.empty()) {
      // TODO: This needs to be adjusted if haze is computed.
      vw_out() << "Exposures exist.";
      return 0;
    }

    // Set up model information
    GlobalParams global_params;
    setUpModelParams(global_params, opt);
    g_reflectance_model_coeffs = &opt.model_coeffs_vec[0];
    
    int num_dems = opt.input_dems.size();

    // Manage no-data
    double dem_nodata_val = -std::numeric_limits<float>::max(); // note we use a float nodata
    if (vw::read_nodata_val(opt.input_dems[0], dem_nodata_val)){
      vw_out() << "Found DEM nodata value: " << dem_nodata_val << std::endl;
      if (std::isnan(dem_nodata_val)) {
        dem_nodata_val = -std::numeric_limits<float>::max(); // bugfix for NaN
        vw_out() << "Overwriting the nodata-value with: " << dem_nodata_val << "\n";
      }
    }
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      double curr_nodata_val = -std::numeric_limits<float>::max(); 
      if (vw::read_nodata_val(opt.input_dems[dem_iter], curr_nodata_val)){
        if (std::isnan(curr_nodata_val)) 
          curr_nodata_val = dem_nodata_val; // bugfix for NaN
        if (dem_nodata_val != curr_nodata_val) {
          vw_throw( ArgumentErr() << "All DEMs must have the same nodata value.\n" );
        }
      }
    }
    if (!boost::math::isnan(opt.nodata_val)) {
      dem_nodata_val = opt.nodata_val;
      vw_out() << "Over-riding the DEM nodata value with: " << dem_nodata_val << std::endl;
    }
    g_dem_nodata_val = &dem_nodata_val;
    
    // Prepare for multiple levels
    int levels = opt.coarse_levels;

    // Read the handles to the DEMs. Here we don't load them into
    // memory yet. We will later load into memory only cropped
    // versions if cropping is specified. This is to save on memory.
    std::vector< ImageViewRef<double> > dem_handles(num_dems);
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) 
      dem_handles[dem_iter] = DiskImageView<double>(opt.input_dems[dem_iter]);

    // There are multiple DEM clips, and multiple coarseness levels
    // for each DEM. Same about albedo and georeferences.
    std::vector< std::vector< ImageView<double> > >
      orig_dems(levels+1), dems(levels+1), albedos(levels+1);
    std::vector<std::vector<GeoReference>> geos(levels+1);
    for (int level = 0; level <= levels; level++) {
      orig_dems [level].resize(num_dems);
      dems      [level].resize(num_dems);
      albedos   [level].resize(num_dems);
      geos      [level].resize(num_dems);
    }
    
    if ( (!opt.crop_win.empty() || opt.query) && num_dems > 1) 
      vw_throw( ArgumentErr() << "Cannot run parallel_stereo with multiple DEM clips.\n" );

    // This must be done before the DEM is cropped. This stats is
    // queried from parallel_sfs.
    if (opt.query) {
      vw_out() << "dem_cols, " << dem_handles[0].cols() << std::endl;
      vw_out() << "dem_rows, " << dem_handles[0].rows() << std::endl;
    }

    // Adjust the crop win
    opt.crop_win.crop(bounding_box(dem_handles[0]));
    
    // Read the georeference. 
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      if (!read_georeference(geos[0][dem_iter], opt.input_dems[dem_iter]))
        vw_throw( ArgumentErr() << "The input DEM has no georeference.\n" );
      
      // Crop the DEM and georef if requested to given box.  The
      // cropped DEM (or uncropped if no cropping happens) is fully
      // loaded in memory.
      if (!opt.crop_win.empty()) {
        dems[0][dem_iter] = crop(dem_handles[dem_iter], opt.crop_win);
        geos[0][dem_iter] = crop(geos[0][dem_iter], opt.crop_win);
      }else{
        dems[0][dem_iter] = dem_handles[dem_iter]; // load in memory
      }
    
      // This can be useful
      vw_out() << "DEM cols and rows: " << dems[0][dem_iter].cols()  << ' '
               << dems[0][dem_iter].rows() << std::endl;
    }
    
    int min_dem_size = 5;
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      // See if to use a constant init value
      if (!boost::math::isnan(opt.init_dem_height)) {
        for (int col = 0; col < dems[0][dem_iter].cols(); col++) {
          for (int row = 0; row < dems[0][dem_iter].rows(); row++) {
            dems[0][dem_iter](col, row) = opt.init_dem_height;
          }
        }
      }

      // Refuse to run if there are no-data values
      for (int col = 0; col < dems[0][dem_iter].cols(); col++) {
        for (int row = 0; row < dems[0][dem_iter].rows(); row++) {
          if (dems[0][dem_iter](col, row) == dem_nodata_val ||
              std::isnan(dems[0][dem_iter](col, row))) {
            vw_throw( ArgumentErr() << "Found a no-data or NaN pixel in the DEM. Cannot continue. "
                      << "The dem_mosaic tool can be used to fill in holes. Then "
                      << "crop and use a clip from this DEM having only valid data.");
          }
        }
      }
      
      if (dems[0][dem_iter].cols() < min_dem_size ||
          dems[0][dem_iter].rows() < min_dem_size) {
        vw_throw( ArgumentErr() << "The input DEM with index "
                  << dem_iter << " is too small.\n" );
      }
    }

    // Read the sun positions from a list, if provided. Usually those
    // are read from the cameras, however, as done further down. 
    std::vector<ModelParams> model_params;
    read_sun_positions_from_list(opt, model_params);
    
    // Read in the camera models (and the sun positions, if not read from the list)
    int num_images = opt.input_images.size();
    std::vector<std::vector<boost::shared_ptr<CameraModel>>> cameras(num_dems);
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {

      cameras[dem_iter].resize(num_images);
      for (int image_iter = 0; image_iter < num_images; image_iter++){
    
        if (opt.skip_images[dem_iter].find(image_iter)
            != opt.skip_images[dem_iter].end()) continue;
    
        typedef boost::scoped_ptr<asp::StereoSession> SessionPtr;
        SessionPtr session(asp::StereoSessionFactory::create
                           (opt.stereo_session, // in-out
                            opt,
                            opt.input_images[image_iter],
                            opt.input_images[image_iter],
                            opt.input_cameras[image_iter],
                            opt.input_cameras[image_iter],
                            opt.out_prefix));
    
        vw_out() << "Loading image and camera: " << opt.input_images[image_iter] << " "
                 <<  opt.input_cameras[image_iter] << " for DEM clip " << dem_iter << ".\n";
        cameras[dem_iter][image_iter] = session->camera_model(opt.input_images[image_iter],
                                                              opt.input_cameras[image_iter]);

        if (dem_iter == 0) {
          // Read the sun position from the camera if it is was not read from the list
          if (model_params[image_iter].sunPosition == Vector3())
            model_params[image_iter].sunPosition
              = sun_position_from_camera(cameras[dem_iter][image_iter]);

          // Sanity check
          if (model_params[image_iter].sunPosition == Vector3())
            vw_throw(ArgumentErr()
                     << "Could not read sun positions from list or from camera model files.\n");
            
          // Compute the azimuth and elevation
          double azimuth, elevation;
          sun_angles(opt, dems[0][dem_iter], dem_nodata_val, geos[0][dem_iter],
                     cameras[dem_iter][image_iter],
                     model_params[image_iter].sunPosition,
                     azimuth, elevation);
          
          // Print this. It will be used to organize the images by illumination
          // for bundle adjustment.
          // Since the sun position has very big values and we want to sort uniquely
          // the images by azimuth angle, use high precision below.
          vw_out().precision(17);
          vw_out() << "Sun position for: " << opt.input_images[image_iter] << " is "
                   << model_params[image_iter].sunPosition << "\n";
          vw_out() << "Sun azimuth and elevation for: "
                   << opt.input_images[image_iter] << " are " << azimuth
                   << " and " << elevation << " degrees.\n";
          vw_out().precision(6); // Go back to usual precision
        }
      }
    }

    // Stop here if all we wanted was some information
    if (opt.query) 
      return 0;

    // This check must be here, after we find the session
    if (opt.stereo_session != "isis" &&
        (opt.use_approx_camera_models || opt.use_approx_adjusted_camera_models ||
         opt.use_rpc_approximation || opt.use_semi_approx)) {
      vw_out() << "Computing approximate models works only with ISIS cameras. "
               << "Ignoring that option.\n";
      opt.use_approx_camera_models = false;
      opt.use_approx_adjusted_camera_models = false;
      opt.use_rpc_approximation = false;
      opt.use_semi_approx = false;
    }
      
    // Since we may float the cameras, ensure our camera models are
    // always adjustable. Note that if the user invoked this tool with
    // --bundle-adjust-prefix, the adjustments were already loaded
    // by now so the cameras are already adjustable. 
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      for (int image_iter = 0; image_iter < num_images; image_iter++){
        
        if (opt.skip_images[dem_iter].find(image_iter) != opt.skip_images[dem_iter].end())
          continue;
        CameraModel * icam
          = dynamic_cast<AdjustedCameraModel*>(cameras[dem_iter][image_iter].get());
        if (icam == NULL) {
          // Set a default identity adjustment
          Vector2 pixel_offset;
          Vector3 translation;
          Quaternion<double> rotation = Quat(math::identity_matrix<3>());
          // For clarity, first make a copy of the object that we will overwrite.
          // This may not be necessary but looks safer this way.
          boost::shared_ptr<CameraModel> cam_ptr = cameras[dem_iter][image_iter];
          cameras[dem_iter][image_iter] = boost::shared_ptr<CameraModel>
            (new AdjustedCameraModel(cam_ptr, translation,
                                     rotation, pixel_offset));
        }
      }
    }
    
    // Prepare for working at multiple levels
    int factor = 2;
    std::vector<int> factors;
    factors.push_back(1);
    for (int level = 1; level <= levels; level++)
      factors.push_back(factors[level-1]*factor);
    
    // We won't load the full images, just portions restricted
    // to the area we we will compute the DEM.
    std::vector<std::vector<std::vector<BBox2i>>> crop_boxes(levels+1);
    for (int level = 0; level <= levels; level++)
      crop_boxes[level].resize(num_dems);
    
    // The crop box starts as the original image bounding box. We'll shrink it later.
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      for (int image_iter = 0; image_iter < num_images; image_iter++){
        std::string img_file = opt.input_images[image_iter];
        crop_boxes[0][dem_iter].push_back(bounding_box(DiskImageView<float>(img_file)));
      }
    }
    
    // Ensure that no two threads can access an ISIS camera at the same time.
    // Declare the lock here, as we want it to live until the end of the program. 
    vw::Mutex camera_mutex;

    // callTop();
    
    // If to use approximate camera models or to crop input images
    if (opt.use_approx_camera_models || opt.use_approx_adjusted_camera_models) {

      // TODO(oalexan1): This code needs to be modularized.
      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      
        double max_approx_err = 0.0;
      
        for (int image_iter = 0; image_iter < num_images; image_iter++){
        
          if (opt.skip_images[dem_iter].find(image_iter)
              != opt.skip_images[dem_iter].end()) continue;
      
          // Here we make a copy, since soon cameras[dem_iter][image_iter] will be overwritten
          AdjustedCameraModel exact_adjusted_camera
            = *dynamic_cast<AdjustedCameraModel*>(cameras[dem_iter][image_iter].get());

          boost::shared_ptr<CameraModel>
            exact_unadjusted_camera = exact_adjusted_camera.unadjusted_model();

          vw_out() << "Creating an approximate camera model for "
                   << opt.input_cameras[image_iter] << " and clip "
                   << opt.input_dems[dem_iter] <<".\n";
          BBox2i img_bbox = crop_boxes[0][dem_iter][image_iter];
          Stopwatch sw;
          sw.start();
          boost::shared_ptr<CameraModel> apcam;
          if (opt.use_approx_camera_models) {
            apcam = boost::shared_ptr<CameraModel>
              (new ApproxCameraModel(exact_adjusted_camera, exact_unadjusted_camera,
                                     img_bbox, dems[0][dem_iter],
                                     geos[0][dem_iter],
                                     dem_nodata_val, opt.use_rpc_approximation,
                                     opt.use_semi_approx,
                                     opt.rpc_penalty_weight, camera_mutex));
            
            // Copy the adjustments over to the approximate camera model
            Vector3 translation  = exact_adjusted_camera.translation();
            Quat rotation        = exact_adjusted_camera.rotation();
            Vector2 pixel_offset = exact_adjusted_camera.pixel_offset();
            double scale         = exact_adjusted_camera.scale();
            cameras[dem_iter][image_iter] = boost::shared_ptr<CameraModel>
              (new AdjustedCameraModel(apcam, translation,
                                       rotation, pixel_offset, scale));
          }else if (opt.use_approx_adjusted_camera_models){
            apcam = boost::shared_ptr<CameraModel>
              (new ApproxAdjustedCameraModel(exact_adjusted_camera, exact_unadjusted_camera,
                                             img_bbox,
                                             dems[0][dem_iter], geos[0][dem_iter],
                                             dem_nodata_val, camera_mutex));
            // Adjustments are already baked into the adjusted
            // approximate cameras, that is why the logic as above to
            // reincorporate the adjustments is not needed.
            cameras[dem_iter][image_iter] = apcam;
          }
          
          sw.stop();
          vw_out() << "Approximate model generation time: " << sw.elapsed_seconds()
                   << " s." << std::endl;
          
          // callTop();

          // Cast the pointer back to ApproxBaseCameraModel as we need that.
          ApproxBaseCameraModel* cam_ptr = dynamic_cast<ApproxBaseCameraModel*>(apcam.get());
          if (cam_ptr == NULL) 
            vw_throw( ArgumentErr() << "Expecting a ApproxBaseCameraModel." );

          bool model_is_valid = cam_ptr->model_is_valid();
          
          // Compared original and unadjusted models
          double max_curr_err = 0.0;

          // TODO: No need to test how unadjusted models compare for RPC,
          // test only the adjusted models. 
          if (model_is_valid) {
            // Recompute the crop box, can be done more reliably here
            if (opt.use_rpc_approximation || opt.use_semi_approx)
              cam_ptr->crop_box() = BBox2();
            for (int col = 0; col < dems[0][dem_iter].cols(); col++) {
              for (int row = 0; row < dems[0][dem_iter].rows(); row++) {
                Vector2 ll = geos[0][dem_iter].pixel_to_lonlat(Vector2(col, row));
                Vector3 xyz = geos[0][dem_iter].datum().geodetic_to_cartesian
                  (Vector3(ll[0], ll[1], dems[0][dem_iter](col, row)));

                if (opt.use_approx_camera_models) {
                  // For approx adjusted camera models we don't do this,
                  // as we don't approximate the unadjusted camera.
                  // Test how unadjusted models compare
                  Vector2 pix1 = exact_unadjusted_camera->point_to_pixel(xyz);
                  //if (!img_bbox.contains(pix1)) continue;
                  
                  Vector2 pix2 = apcam->point_to_pixel(xyz);
                  max_curr_err = std::max(max_curr_err, norm_2(pix1 - pix2));
                  
                  // Use these pixels to expand the crop box, as we
                  // now also know the adjustments.  This is a bug
                  // fix.
                  cam_ptr->crop_box().grow(pix1);
                  cam_ptr->crop_box().grow(pix2);
                }
                
                // Test how adjusted (exact and approximate) models compare
                Vector2 pix3 = exact_adjusted_camera.point_to_pixel(xyz);
                //if (!img_bbox.contains(pix3)) continue;
                Vector2 pix4 = cameras[dem_iter][image_iter]->point_to_pixel(xyz);
                max_curr_err = std::max(max_curr_err, norm_2(pix3 - pix4));

                cam_ptr->crop_box().grow(pix3);
                cam_ptr->crop_box().grow(pix4);
              }
            }

            cam_ptr->crop_box().crop(img_bbox);
            
            vw_out() << "Max approximate model error in pixels for: "
                     <<  opt.input_images[image_iter] << " and clip "
                     << opt.input_dems[dem_iter] << ": " << max_curr_err << std::endl;
          }else{
            vw_out() << "Invalid model for clip: " << dem_iter << ".\n";
          }
          
          if (max_curr_err > opt.rpc_max_error || !model_is_valid) {
            // This is a bugfix. When the DEM clip does not intersect the image,
            // the approx camera model has incorrect values.
            if (model_is_valid)
              vw_out() << "Error is too big.\n";
            vw_out() << "Skip image " << image_iter << " for clip "
                     << dem_iter << std::endl;
            opt.skip_images[dem_iter].insert(image_iter);
            cam_ptr->crop_box() = BBox2();
            max_curr_err = 0.0;
          }

          max_approx_err = std::max(max_approx_err, max_curr_err);
        
          if (opt.use_rpc_approximation && !cam_ptr->crop_box().empty()){
            // Grow the box just a bit more, to ensure we still see
            // enough of the images during optimization.
            double extra = 0.2;
            double extrax = extra*cam_ptr->crop_box().width();
            double extray = extra*cam_ptr->crop_box().height();
            cam_ptr->crop_box().min() -= Vector2(extrax, extray);
            cam_ptr->crop_box().max() += Vector2(extrax, extray);
          }
          cam_ptr->crop_box().crop(img_bbox);
          vw_out() << "Crop box dimensions: " << cam_ptr->crop_box() << std::endl;
    
          // Copy the crop box
          if (opt.crop_input_images)
            crop_boxes[0][dem_iter][image_iter].crop(cam_ptr->crop_box());

          // Skip images which result in empty crop boxes
          if (crop_boxes[0][dem_iter][image_iter].empty()) {
            opt.skip_images[dem_iter].insert(image_iter);
          }
          
        } // end iterating over images
        vw_out() << "Max total approximate model error in pixels: "
                 << max_approx_err << std::endl;
        
      } // end iterating over dem clips

      // end computing the approximate camera model
    } else if (opt.crop_input_images) {
      
      // We will arrive here if it is desired to crop the input images
      // without using an approximate model, such as for CSM.
      // Estimate the crop box by projecting the pixels in the exact
      // camera (with the adjustments applied, if present).
      
      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
        for (int image_iter = 0; image_iter < num_images; image_iter++){
          if (opt.skip_images[dem_iter].find(image_iter)
              != opt.skip_images[dem_iter].end()) continue;

          // Store the full image box, and initialize the crop box to an empty box
          BBox2i img_bbox = crop_boxes[0][dem_iter][image_iter];
          crop_boxes[0][dem_iter][image_iter] = BBox2i();
          
          for (int col = 0; col < dems[0][dem_iter].cols(); col++) {
            for (int row = 0; row < dems[0][dem_iter].rows(); row++) {
              Vector2 ll = geos[0][dem_iter].pixel_to_lonlat(Vector2(col, row));
              Vector3 xyz = geos[0][dem_iter].datum().geodetic_to_cartesian
                (Vector3(ll[0], ll[1], dems[0][dem_iter](col, row)));
              
              Vector2 pix = cameras[dem_iter][image_iter]->point_to_pixel(xyz);
              crop_boxes[0][dem_iter][image_iter].grow(pix); 
            }
          }

          // Double the box dimensions, just in case. Later the SfS heights
          // may change, and we may need to see beyond the given box
          double extraFactor = 0.5;
          double extrax = extraFactor * crop_boxes[0][dem_iter][image_iter].width();
          double extray = extraFactor * crop_boxes[0][dem_iter][image_iter].height();
          crop_boxes[0][dem_iter][image_iter].min() -= Vector2(extrax, extray);
          crop_boxes[0][dem_iter][image_iter].max() += Vector2(extrax, extray);

          // Crop to the bounding box of the image
          crop_boxes[0][dem_iter][image_iter].crop(img_bbox);
            
          vw_out() << "Estimated crop box for image " 
                   << opt.input_images[image_iter] << " and clip "
                   << opt.input_dems[dem_iter] << ": "
                   << crop_boxes[0][dem_iter][image_iter]
                   << std::endl;
          
          if (crop_boxes[0][dem_iter][image_iter].empty()) 
            opt.skip_images[dem_iter].insert(image_iter);
        }
      }
    }

    // Compute the boxes at lower resolutions
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      
      // Make the crop boxes lower left corner be multiple of 2^level
      int last_factor = factors.back();
      for (int image_iter = 0; image_iter < num_images; image_iter++){
        if (!crop_boxes[0][dem_iter][image_iter].empty()) {
          Vector2i mn = crop_boxes[0][dem_iter][image_iter].min();
          crop_boxes[0][dem_iter][image_iter].min()
            = last_factor*(floor(mn/double(last_factor)));
        }
      }
      
      // Crop boxes at the coarser resolutions
      for (int image_iter = 0; image_iter < num_images; image_iter++){
        for (int level = 1; level <= levels; level++) {
          crop_boxes[level][dem_iter]
            .push_back(crop_boxes[0][dem_iter][image_iter]/factors[level]);
        }
      }
    }
    
    // Masked images and weights.
    std::vector<std::vector< std::vector<MaskedImgT> > > masked_images_vec(levels+1);
    std::vector<std::vector< std::vector<DoubleImgT> > > blend_weights_vec(levels+1);
    for (int level = levels; level >= 0; level--) {
      masked_images_vec[level].resize(num_dems);
      blend_weights_vec[level].resize(num_dems);
      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
        masked_images_vec[level][dem_iter].resize(num_images);
        blend_weights_vec[level][dem_iter].resize(num_images);
      }
    }
    
    float img_nodata_val = -std::numeric_limits<float>::max();
    for (int image_iter = 0; image_iter < num_images; image_iter++){
      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      
        if (opt.skip_images[dem_iter].find(image_iter) != opt.skip_images[dem_iter].end())
          continue;
       
        std::string img_file = opt.input_images[image_iter];
        if (vw::read_nodata_val(img_file, img_nodata_val)){
          //vw_out() << "Found image " << image_iter << " nodata value: "
          //         << img_nodata_val << std::endl;
        }
        // Model the shadow threshold
        float shadow_thresh = opt.shadow_threshold_vec[image_iter];
        if (opt.crop_input_images) {
          // Make a copy in memory for faster access
          if (!crop_boxes[0][dem_iter][image_iter].empty()) {
            ImageView<float> cropped_img = 
              crop(DiskImageView<float>(img_file), crop_boxes[0][dem_iter][image_iter]);
            masked_images_vec[0][dem_iter][image_iter]
              = create_pixel_range_mask2(cropped_img,
                                         std::max(img_nodata_val, shadow_thresh),
                                         opt.max_valid_image_vals_vec[image_iter]);

            // Compute blending weights only when cropping the
            // images. Otherwise the weights are too huge.
            if (opt.blending_dist > 0)
              blend_weights_vec[0][dem_iter][image_iter]
                = asp::blendingWeights(masked_images_vec[0][dem_iter][image_iter],
                                       opt.blending_dist, opt.blending_power,
                                       opt.min_blend_size);
          }
        }else{
          masked_images_vec[0][dem_iter][image_iter]
            = create_pixel_range_mask2(DiskImageView<float>(img_file),
                                       std::max(img_nodata_val, shadow_thresh),
                                       opt.max_valid_image_vals_vec[image_iter]);
        }
      }
    }
    g_img_nodata_val = &img_nodata_val;

    // Copy sun positions to an array
    std::vector<double> scaled_sun_posns(3*num_images);
    for (int image_iter = 0; image_iter < num_images; image_iter++){
      for (int it = 0; it < 3; it++) 
        scaled_sun_posns[3*image_iter + it] = 1; // model_params[image_iter].sunPosition[it];
    }    
    
    // Find the grid sizes in meters. Note that dem heights are in
    // meters too, so we treat both horizontal and vertical
    // measurements in same units.
    double gridx, gridy;
    compute_grid_sizes_in_meters(dems[0][0], geos[0][0], dem_nodata_val, gridx, gridy);
    vw_out() << "grid in x and y in meters: "
             << gridx << ' ' << gridy << std::endl;
    g_gridx = &gridx;
    g_gridy = &gridy;

    // Find the max DEM height
    std::vector<double> max_dem_height(num_dems, -std::numeric_limits<double>::max());
    if (opt.model_shadows) {
      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
        double curr_max_dem_height = -std::numeric_limits<double>::max();
        for (int col = 0; col < dems[0][dem_iter].cols(); col++) {
          for (int row = 0; row < dems[0][dem_iter].rows(); row++) {
            if (dems[0][dem_iter](col, row) > curr_max_dem_height) {
              curr_max_dem_height = dems[0][dem_iter](col, row);
            }
          }
        }
        max_dem_height[dem_iter] = curr_max_dem_height;
      }
    }
    g_max_dem_height = &max_dem_height;
    
    // Initial albedo. This will be updated later.
    double initial_albedo = 1.0;
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      albedos[0][dem_iter].set_size(dems[0][dem_iter].cols(), dems[0][dem_iter].rows());
      for (int col = 0; col < albedos[0][dem_iter].cols(); col++) {
        for (int row = 0; row < albedos[0][dem_iter].rows(); row++) {
          albedos[0][dem_iter](col, row) = initial_albedo;
        }
      }
    }
    
    // We have intensity = albedo * nonlin_reflectance(reflectance, exposure, haze, num_haze_coeffs)
    // Assume that haze is 0 to start with. Find the exposure as
    // mean(intensity)/mean(reflectance)/albedo. Use this to compute an
    // initial exposure and decide based on that which images to
    // skip. If the user provided initial exposures and haze, use those, but
    // still go through the motions to find the images to skip.
    vw_out() << "Computing exposures.\n";
    std::vector<double> local_exposures_vec(num_images, 0);
    for (int image_iter = 0; image_iter < num_images; image_iter++) {
      
      std::vector<double> exposures_per_dem;
      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
        
        if (opt.skip_images[dem_iter].find(image_iter) !=
            opt.skip_images[dem_iter].end()) continue;
        
        ImageView<PixelMask<double>> reflectance, intensity;
        ImageView<double> ground_weight;
        ImageView<Vector2> pq; // no need for these just for initialization
        
        // Sample the large DEMs. Keep about 200 row and column samples.
        int sample_col_rate = std::max((int)round(dems[0][dem_iter].cols()/200.0), 1);
        int sample_row_rate = std::max((int)round(dems[0][dem_iter].rows()/200.0), 1);
        computeReflectanceAndIntensity(dems[0][dem_iter], pq, geos[0][dem_iter],
                                       opt.model_shadows, max_dem_height[dem_iter],
                                       gridx, gridy, sample_col_rate, sample_row_rate,
                                       model_params[image_iter],
                                       global_params,
                                       crop_boxes[0][dem_iter][image_iter],
                                       masked_images_vec[0][dem_iter][image_iter],
                                       blend_weights_vec[0][dem_iter][image_iter],
                                       cameras[dem_iter][image_iter].get(),
                                       &scaled_sun_posns[3*image_iter],
                                       reflectance, intensity, ground_weight,
                                       &opt.model_coeffs_vec[0]);
        
        // TODO: Below is not the optimal way of finding the exposure!
        // Find it as the analytical minimum using calculus.
        double imgmean, imgstdev, refmean, refstdev;
        compute_image_stats(intensity, reflectance, imgmean, imgstdev, refmean, refstdev);
        double exposure = imgmean/refmean/initial_albedo;
        vw_out() << "img mean std: " << imgmean << ' ' << imgstdev << std::endl;
        vw_out() << "ref mean std: " << refmean << ' ' << refstdev << std::endl;
        vw_out() << "Local exposure for image " << image_iter << " and clip "
                 << dem_iter << ": " << exposure << std::endl;
    
        double big = 1e+100; // There's no way image exposure can be bigger than this
        bool is_good = ( 0 < exposure && exposure < big );
        if (is_good) {
          exposures_per_dem.push_back(exposure);
        }else{
          // Skip images with bad exposure. Apparently there is no good
          // imagery in the area.
          opt.skip_images[dem_iter].insert(image_iter);
          vw_out() << "Skip image " << image_iter << " for clip " << dem_iter << std::endl;
        }
      }
      
      // Out the exposures for this image on all clips, pick the median
      int len = exposures_per_dem.size();
      if (len > 0) {
        std::sort(exposures_per_dem.begin(), exposures_per_dem.end());
        local_exposures_vec[image_iter] = 
          0.5*(exposures_per_dem[(len-1)/2] + exposures_per_dem[len/2]);
        //vw_out() << "Median exposure for image " << image_iter << " on all clips: "
        //     << local_exposures_vec[image_iter] << std::endl;
      }
      
    }
    
    // Only overwrite the exposures if we don't have them supplied
    if (opt.image_exposures_vec.empty()) opt.image_exposures_vec = local_exposures_vec;

    for (size_t image_iter = 0; image_iter < opt.image_exposures_vec.size(); image_iter++) {
      vw_out() << "Image exposure for " << opt.input_images[image_iter] << ' '
               << opt.image_exposures_vec[image_iter] << std::endl;
    }

    // Initialize the haze as 0.
    if ((!opt.image_haze_vec.empty()) && (int)opt.image_haze_vec.size() != num_images)
      vw_throw(ArgumentErr() << "Expecting as many haze values as images.\n");
    if (opt.image_haze_vec.empty()) {
      for (int image_iter = 0; image_iter < num_images; image_iter++) {
        // Pad the haze vec
        std::vector<double> haze_vec;
        while (haze_vec.size() < g_max_num_haze_coeffs) haze_vec.push_back(0);
        opt.image_haze_vec.push_back(haze_vec);
      }
    }
    if (opt.compute_exposures_only) {
      save_exposures(opt.out_prefix, opt.input_images, opt.image_exposures_vec);
      // all done
      return 0;
    }

    // Need to compute the valid data image to be able to find the grid points always
    // in shadow, so when this image is zero.
    ImageView<int> lit_image_mask;
    if (opt.curvature_in_shadow_weight > 0.0) {
      if (num_dems > 1 || levels > 0) 
        vw_throw(ArgumentErr() << "Enforcing positive curvature in shadow does not work "
                 << "with more than one input DEM clip or a positive number of "
                 << "coarseness levels.\n");
      lit_image_mask.set_size(dems[0][0].cols(), dems[0][0].rows());
      for (int col = 0; col < lit_image_mask.cols(); col++) {
        for (int row = 0; row < lit_image_mask.rows(); row++) {
          lit_image_mask(col, row) = 0; // no valid points originally
        }
      }
    }

    // If opt.allow_borderline_data is true, create for each image that will not be skipped
    // a weight matrix with dimensions equal to DEM dimensions, that will be used instead
    // of weights in the camera image space. These are balanced among each other and give more
    // weight to barely lit and unlit nearby pixels.
    std::vector<ImageView<double>> ground_weights(num_images);
    
    // Note that below we may use the exposures computed at the previous step
    if (opt.save_computed_intensity_only || opt.estimate_slope_errors ||
        opt.estimate_height_errors || opt.curvature_in_shadow_weight > 0.0 ||
        opt.allow_borderline_data) {
      // In this case simply save the computed and actual intensity, and for most of these quit
      ImageView<PixelMask<double>> reflectance, meas_intensity, comp_intensity;
      ImageView<double> ground_weight;
      ImageView<Vector2> pq; // no need for these just for initialization
      int sample_col_rate = 1, sample_row_rate = 1;

      boost::shared_ptr<SlopeErrEstim> slopeErrEstim = boost::shared_ptr<SlopeErrEstim>(NULL);
      if (opt.estimate_slope_errors) {
        int num_a_samples = 90; // Sample the 0 to 90 degree range with this many samples
        int num_b_samples = 360; // sample the 0 to 360 degree range with this many samples
        slopeErrEstim = boost::shared_ptr<SlopeErrEstim>
          (new SlopeErrEstim(dems[0][0].cols(), dems[0][0].rows(),
                             num_a_samples, num_b_samples, &albedos[0][0], &opt));
      }
      
      boost::shared_ptr<HeightErrEstim> heightErrEstim = boost::shared_ptr<HeightErrEstim>(NULL);
      if (opt.estimate_height_errors) {
        double max_height_error  = opt.height_error_params[0];
        int num_height_samples   = opt.height_error_params[1];
        vw_out() << "Maximum height error to examine: " << max_height_error << "\n";
        vw_out() << "Number of samples to use from 0 to that height: " << num_height_samples
                 << "\n";
          
        double nodata_height_val = -1.0;
        heightErrEstim = boost::shared_ptr<HeightErrEstim>
          (new HeightErrEstim(dems[0][0].cols(), dems[0][0].rows(),
                              num_height_samples, max_height_error, nodata_height_val,
                              &albedos[0][0], &opt));
      }
      
      for (int image_iter = 0; image_iter < num_images; image_iter++) {
        
        if (opt.estimate_slope_errors) 
          slopeErrEstim->image_iter = image_iter;
        if (opt.estimate_height_errors) 
          heightErrEstim->image_iter = image_iter;

        // Find the reflectance and measured intensity (and work towards estimating the slopes
        // if asked to).
        computeReflectanceAndIntensity(dems[0][0], pq, geos[0][0],
                                       opt.model_shadows, max_dem_height[0],
                                       gridx, gridy, sample_col_rate, sample_row_rate,
                                       model_params[image_iter],
                                       global_params,
                                       crop_boxes[0][0][image_iter],
                                       masked_images_vec[0][0][image_iter],
                                       blend_weights_vec[0][0][image_iter],
                                       cameras[0][image_iter].get(),
                                       &scaled_sun_posns[3*image_iter],
                                       reflectance, meas_intensity, ground_weight,
                                       &opt.model_coeffs_vec[0],
                                       slopeErrEstim.get(), heightErrEstim.get());

        if (opt.skip_images[0].find(image_iter) == opt.skip_images[0].end() &&
            opt.allow_borderline_data) {
          // if not skipping, save the weight
          ground_weights[image_iter] = copy(ground_weight);
        }
        
        // Find the computed intensity.
        // TODO(oalexan1): Should one mark the no-data values rather than setting
        // them to 0? 
        comp_intensity.set_size(reflectance.cols(), reflectance.rows());
        for (int col = 0; col < comp_intensity.cols(); col++) {
          for (int row = 0; row < comp_intensity.rows(); row++) {
            comp_intensity(col, row)
              = albedos[0][0](col, row) *
              nonlin_reflectance(reflectance(col, row), opt.image_exposures_vec[image_iter],
                                 opt.steepness_factor,
                                 &opt.image_haze_vec[image_iter][0], opt.num_haze_coeffs);
          }
        }
        
        if (opt.curvature_in_shadow_weight > 0.0) {
          if (meas_intensity.cols() != lit_image_mask.cols() ||
              meas_intensity.rows() != lit_image_mask.rows()) {
            vw_throw(ArgumentErr()
                     << "Intensity image dimensions disagree with DEM clip dimensions.\n");
          }
          for (int col = 0; col < lit_image_mask.cols(); col++) {
            for (int row = 0; row < lit_image_mask.rows(); row++) {
              if (is_valid(meas_intensity(col, row))           || 
                  col == 0 || col == lit_image_mask.cols() - 1 ||
                  row == 0 || row == lit_image_mask.rows() - 1) {
                // Boundary pixels are declared lit. Otherwise they are
                // always unlit due to the peculiarities of how the intensity
                // is found at the boundary.
                lit_image_mask(col, row) = 1;
              }
            }
          }
        }
        
        if (opt.save_computed_intensity_only) {
          TerminalProgressCallback tpc("asp", ": ");
          bool has_georef = true, has_nodata = true;
          std::string out_camera_file
            = asp::bundle_adjust_file_name(opt.out_prefix,
                                           opt.input_images[image_iter],
                                           opt.input_cameras[image_iter]);
          std::string local_prefix = fs::path(out_camera_file).replace_extension("").string();
          std::string out_meas_intensity_file = local_prefix + "-meas-intensity.tif";
          vw_out() << "Writing: " << out_meas_intensity_file << std::endl;
          block_write_gdal_image(out_meas_intensity_file,
                                 apply_mask(meas_intensity, img_nodata_val),
                                 has_georef, geos[0][0], has_nodata,
                                 img_nodata_val, opt, tpc);
          
          std::string out_comp_intensity_file = local_prefix + "-comp-intensity.tif";
          vw_out() << "Writing: " << out_comp_intensity_file << std::endl;
          block_write_gdal_image(out_comp_intensity_file,
                                 apply_mask(comp_intensity, img_nodata_val),
                                 has_georef, geos[0][0], has_nodata, img_nodata_val,
                                 opt, tpc);
        }

      } // End iterating over images

      if (opt.estimate_slope_errors) {
        // Find the slope error as the maximum of slope errors in all directions
        // from the given slope.
        ImageView<float> slope_error;
        slope_error.set_size(reflectance.cols(), reflectance.rows());
        double nodata_slope_value = -1.0;
        for (int col = 0; col < slope_error.cols(); col++) {
          for (int row = 0; row < slope_error.rows(); row++) {
            slope_error(col, row) = nodata_slope_value;
            int num_samples = slopeErrEstim->slope_errs[col][row].size();
            for (int sample = 0; sample < num_samples; sample++) {
              slope_error(col, row)
                = std::max(double(slope_error(col, row)),
                           slopeErrEstim->slope_errs[col][row][sample]);
            }
          }
        }
        
        // Slope errors that are stuck at 90 degrees could not be estimated
        for (int col = 0; col < slope_error.cols(); col++) {
          for (int row = 0; row < slope_error.rows(); row++) {
            if (slope_error(col, row) == slopeErrEstim->max_angle)
              slope_error(col, row) = nodata_slope_value;
          }
        }
        
        TerminalProgressCallback tpc("asp", ": ");
        bool has_georef = true, has_nodata = true;
        std::string slope_error_file = opt.out_prefix + "-slope-error.tif";
        vw_out() << "Writing: " << slope_error_file << std::endl;
        block_write_gdal_image(slope_error_file,
                               slope_error, has_georef, geos[0][0], has_nodata,
                               nodata_slope_value, opt, tpc);
      }

      if (opt.estimate_height_errors) {
        // Find the height error from the range of heights
        ImageView<float> height_error;
        height_error.set_size(heightErrEstim->height_error_vec.cols(),
                              heightErrEstim->height_error_vec.rows());
        for (int col = 0; col < height_error.cols(); col++) {
          for (int row = 0; row < height_error.rows(); row++) {
            height_error(col, row)
              = std::max(-heightErrEstim->height_error_vec(col, row)[0],
                         heightErrEstim->height_error_vec(col, row)[1]);
            
            // When we are stuck at the highest error that means we could not
            // find it
            if (height_error(col, row) == heightErrEstim->max_height_error)
              height_error(col, row) = heightErrEstim->nodata_height_val;
          }
        }
        TerminalProgressCallback tpc("asp", ": ");
        bool has_georef = true, has_nodata = true;
        std::string height_error_file = opt.out_prefix + "-height-error.tif";
        vw_out() << "Writing: " << height_error_file << std::endl;
        block_write_gdal_image(height_error_file,
                               height_error,
                               has_georef, geos[0][0],
                               has_nodata, heightErrEstim->nodata_height_val,
                               opt, tpc);
      }
      
    } // End doing intensity computations and/or height and/or slope error estimations
      
    if (opt.save_computed_intensity_only || opt.estimate_slope_errors ||
        opt.estimate_height_errors) {
      save_exposures(opt.out_prefix, opt.input_images, opt.image_exposures_vec);
      // All done
      return 0;
    }

    if (opt.allow_borderline_data) {
      int cols = dems[0][0].cols(), rows = dems[0][0].rows();
      asp::adjustBorderlineDataWeights(cols, rows, opt.blending_dist, opt.blending_power,
                                       vw::GdalWriteOptions(opt), // slice
                                       geos[0][0],
                                       opt.skip_images[0],
                                       opt.out_prefix, // for debug data
                                       opt.input_images, opt.input_cameras, 
                                       ground_weights); // output

      // Use the ground weights from now on instead of blending weights.
      // Will overwrite the weights below.
      g_blend_weight_is_ground_weight = true;
      if (num_dems != 1) 
        vw::vw_throw(vw::ArgumentErr() << "Cannot use more than one DEM with "
                     << "--allow-borderline-data.\n");

      // Redo the image masks. Unlike before, the shadow threshold is set to 0
      // to allow shadow pixels. The weights will control how much of these
      // are actually used. This approach is better than a hard cutoff with the mask.
      for (int image_iter = 0; image_iter < num_images; image_iter++) {
        for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
          if (opt.skip_images[dem_iter].find(image_iter) != opt.skip_images[dem_iter].end())
            continue;
          
          std::string img_file = opt.input_images[image_iter];
          vw::read_nodata_val(img_file, img_nodata_val);
          float shadow_thresh = 0.0; // Note how the shadow thresh is now 0, unlike before
          // Make a copy in memory for faster access
          if (!crop_boxes[0][dem_iter][image_iter].empty()) {
            ImageView<float> cropped_img = 
              crop(DiskImageView<float>(img_file), crop_boxes[0][dem_iter][image_iter]);
            masked_images_vec[0][dem_iter][image_iter]
              = create_pixel_range_mask2(cropped_img,
                                         std::max(img_nodata_val, shadow_thresh),
                                         opt.max_valid_image_vals_vec[image_iter]);

            // Overwrite the blending weights with ground weights
            blend_weights_vec[0][dem_iter][image_iter] = copy(ground_weights[image_iter]);
          }
        }
      }

      ground_weights.clear(); // not needed anymore
    } // end allow borderline data
    
    ImageView<double> curvature_in_shadow_weight;
    if (opt.curvature_in_shadow_weight > 0.0) {
      TerminalProgressCallback tpc("asp", ": ");
      bool has_georef = true, has_nodata = false;
      double nodata_val = -1; // will not be used
      std::string lit_image_mask_file = opt.out_prefix + "-lit_image_mask.tif";
      vw_out() << "Writing: " << lit_image_mask_file << std::endl;
      block_write_gdal_image(lit_image_mask_file, lit_image_mask,
                             has_georef, geos[0][0], has_nodata, nodata_val, opt, tpc);

      // Form the curvature_in_shadow_weight image. It will start at 0
      // at distance opt.lit_curvature_dist from the shadow
      // boundary in the lit area, and then reach value
      // opt.curvature_in_shadow_weight when at distance
      // opt.shadow_curvature_dist from the boundary in the shadowed
      // area. This is done to avoid boundary artifacts.
      double max_dist = std::max(opt.lit_curvature_dist, opt.shadow_curvature_dist);
      vw::bounded_signed_dist<int>(vw::create_mask(lit_image_mask, 0), max_dist,
                                   curvature_in_shadow_weight);
      // Do further adjustments
      for (int col = 0; col < curvature_in_shadow_weight.cols(); col++) {
        for (int row = 0; row < curvature_in_shadow_weight.rows(); row++) {
          double val = curvature_in_shadow_weight(col, row);
          val = std::min(val, opt.lit_curvature_dist);
          val = std::max(val, -opt.shadow_curvature_dist);
          val = (opt.lit_curvature_dist - val) /
            (opt.lit_curvature_dist + opt.shadow_curvature_dist);
          curvature_in_shadow_weight(col, row) = val * opt.curvature_in_shadow_weight;
        }
      }
      
      std::string curvature_in_shadow_weight_file = opt.out_prefix
        + "-curvature_in_shadow_weight.tif";
      vw_out() << "Writing: " << curvature_in_shadow_weight_file << std::endl;
      block_write_gdal_image(curvature_in_shadow_weight_file, curvature_in_shadow_weight,
                             has_georef, geos[0][0], has_nodata, nodata_val, opt, tpc);
    }
    
    if (opt.num_haze_coeffs > 0) {
      for (size_t image_iter = 0; image_iter < opt.image_haze_vec.size(); image_iter++) {
        vw_out() << "Image haze for " << opt.input_images[image_iter] << ':';
        for (size_t hiter = 0; hiter < opt.image_haze_vec[image_iter].size(); hiter++) {
          vw_out() << " " << opt.image_haze_vec[image_iter][hiter];
        }
        vw_out() << "\n";
      }
    }
    
    g_exposures        = &opt.image_exposures_vec;
    g_haze             = &opt.image_haze_vec;
    g_scaled_sun_posns = &scaled_sun_posns;
    
    // For images that we don't use, wipe the cameras and all other
    // info, as those take up memory (the camera is a table). 
    for (int image_iter = 0; image_iter < num_images; image_iter++) {
      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
        if (opt.skip_images[dem_iter].find(image_iter) != opt.skip_images[dem_iter].end()) {
          masked_images_vec[0][dem_iter][image_iter] = ImageView<PixelMask<float>>();
          blend_weights_vec[0][dem_iter][image_iter] = ImageView<double>();
          cameras[dem_iter][image_iter] = boost::shared_ptr<CameraModel>();
        }
      }
    }
    
    // The initial camera adjustments. They will be updated later.
    std::vector<double> adjustments(6*num_images, 0);
    for (int image_iter = 0; image_iter < num_images; image_iter++) {

      for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
        if (opt.skip_images[dem_iter].find(image_iter) != opt.skip_images[dem_iter].end()) continue;
      
        Vector3 translation, axis_angle;
        Vector2 pixel_offset;

        if (!opt.use_approx_adjusted_camera_models) {
          AdjustedCameraModel * icam
            = dynamic_cast<AdjustedCameraModel*>(cameras[dem_iter][image_iter].get());
          if (icam == NULL)
            vw_throw(ArgumentErr() << "Expecting an adjusted camera model.\n");
          translation = icam->translation();
          axis_angle = icam->rotation().axis_angle();
          pixel_offset = icam->pixel_offset();
        }else{
          ApproxAdjustedCameraModel * aapcam
            = dynamic_cast<ApproxAdjustedCameraModel*>(cameras[dem_iter][image_iter].get());
          if (aapcam == NULL)
            vw_throw(ArgumentErr() << "Expecting an approximate adjusted camera model.\n");
          AdjustedCameraModel acam = aapcam->exact_adjusted_camera();
          translation = acam.translation();
          axis_angle = acam.rotation().axis_angle();
          pixel_offset = acam.pixel_offset();
        }

        // TODO(oalexan1): This does not appear necessary use adjusted approximate cameras.
        if (pixel_offset != Vector2())
          vw_throw(ArgumentErr() << "Expecting zero pixel offset.\n");
        for (int param_iter = 0; param_iter < 3; param_iter++) {
          adjustments[6*image_iter + 0 + param_iter]
            = translation[param_iter]/(g_position_scale_factor*opt.camera_position_step_size);
          adjustments[6*image_iter + 3 + param_iter] = axis_angle[param_iter];
        }
      }
    }
    g_adjustments = &adjustments;

    // Prepare data at each coarseness level
    // orig_dems will keep the input DEMs and won't change. Keep to the optimized
    // DEMs close to orig_dems. Make a deep copy below.
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
      orig_dems[0][dem_iter] = copy(dems[0][dem_iter]);
    }
    
    double sub_scale = 1.0/factor;
    for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {

      for (int level = 1; level <= levels; level++) {
        geos[level][dem_iter] = resample(geos[level-1][dem_iter], sub_scale);
        orig_dems[level][dem_iter]
          = pixel_cast<double>(vw::resample_aa
                               (pixel_cast< PixelMask<double> >
                                (orig_dems[level-1][dem_iter]), sub_scale));
        dems[level][dem_iter] = copy(orig_dems[level][dem_iter]);
        
        // CERES won't be happy with tiny DEMs
        if (dems[level][dem_iter].cols() < min_dem_size || dems[level][dem_iter].rows()
            < min_dem_size) {
          levels = std::max(0, level-1);
          vw_out(WarningMessage) << "Reducing the number of coarse levels to "
                                 << levels << ".\n";
          geos.resize(levels+1);
          orig_dems.resize(levels+1);
          dems.resize(levels+1);
          albedos.resize(levels+1);
          masked_images_vec.resize(levels+1);
          blend_weights_vec.resize(levels+1);
          factors.resize(levels+1);
          break;
        }

        albedos[level][dem_iter]
          = pixel_cast<double>(vw::resample_aa
                               (pixel_cast< PixelMask<double>>
                                (albedos[level-1][dem_iter]), sub_scale));

        // We must write the subsampled images to disk, and then read
        // them back, as VW cannot access individual pixels of the
        // monstrosities created using the logic below, and even if it
        // could, it is best if resampling is done once, and offline,
        // rather than redoing it each time within the optimization
        // loop.
        for (int image_iter = 0; image_iter < num_images; image_iter++) {

          if (opt.skip_images[dem_iter].find(image_iter)
              != opt.skip_images[dem_iter].end()) continue;
        
          fs::path image_path(opt.input_images[image_iter]);
          std::ostringstream os; os << "-level" << level;
          if (num_dems > 1)      os << "-clip"  << dem_iter;
          std::string sub_image = opt.out_prefix + "-"
            + image_path.stem().string() + os.str() + ".tif";
          vw_out() << "Writing subsampled image: " << sub_image << "\n";
          bool has_img_georef = false;
          GeoReference img_georef;
          bool has_img_nodata = true;
          int tile_size = 256;
          int sub_threads = 1;
          TerminalProgressCallback tpc("asp", ": ");
          vw::cartography::block_write_gdal_image
            (sub_image,
             apply_mask
             (block_rasterize
              (vw::cache_tile_aware_render
               (vw::resample_aa
                (masked_images_vec[level-1][dem_iter][image_iter], sub_scale),
                Vector2i(tile_size, tile_size) * sub_scale),
               Vector2i(tile_size, tile_size), sub_threads), img_nodata_val),
             has_img_georef, img_georef, has_img_nodata, img_nodata_val, opt, tpc);
          
          // Read it right back
          if (opt.crop_input_images) {
            // Read it fully in memory, as we cropped it before
            ImageView<float> memory_img = copy(DiskImageView<float>(sub_image));
            masked_images_vec[level][dem_iter][image_iter]
              = create_mask(memory_img, img_nodata_val);
          }else{
            // Read just a handle, as the full image could be huge
            masked_images_vec[level][dem_iter][image_iter]
              = create_mask(DiskImageView<float>(sub_image), img_nodata_val);
          }
        
          if (blend_weights_vec[level-1][dem_iter][image_iter].cols() > 0 &&
              blend_weights_vec[level-1][dem_iter][image_iter].rows() > 0 ) {
            fs::path weight_path(opt.input_images[image_iter]);
            std::string sub_weight = opt.out_prefix + "-wt-"
              + weight_path.stem().string() + os.str() + ".tif";
            vw_out() << "Writing subsampled weight: " << sub_weight << "\n";
          
            vw::cartography::block_write_gdal_image
              (sub_weight,
               apply_mask
               (block_rasterize
                (vw::cache_tile_aware_render
                 (vw::resample_aa
                  (create_mask(blend_weights_vec[level-1][dem_iter][image_iter],
                               dem_nodata_val), sub_scale),
                  Vector2i(tile_size,tile_size) * sub_scale),
                 Vector2i(tile_size, tile_size), sub_threads), dem_nodata_val),
               has_img_georef, img_georef, has_img_nodata, dem_nodata_val, opt, tpc);

            ImageView<double> memory_weight = copy(DiskImageView<double>(sub_weight));
            blend_weights_vec[level][dem_iter][image_iter] = memory_weight;
          }
        
        }
      }
    }
    
    // Start going from the coarsest to the finest level
    for (int level = levels; level >= 0; level--) {

      g_level = level;

      int num_iterations;
      if (level == 0)
        num_iterations = opt.max_iterations;
      else
        num_iterations = opt.max_coarse_iterations;

      // Scale the cameras
      for (int image_iter = 0; image_iter < num_images; image_iter++) {
        for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
          
          if (opt.skip_images[dem_iter].find(image_iter) !=
              opt.skip_images[dem_iter].end()) continue;

          if (!opt.use_approx_adjusted_camera_models) {
            AdjustedCameraModel * adj_cam
              = dynamic_cast<AdjustedCameraModel*>(cameras[dem_iter][image_iter].get());
            if (adj_cam == NULL)
              vw_throw( ArgumentErr() << "Expecting adjusted camera.\n");
            adj_cam->set_scale(factors[level]);
          }
        }
      }
      
      run_sfs_level(// Fixed inputs
                    num_iterations, opt, geos[level],
                    opt.smoothness_weight*factors[level]*factors[level],
                    dem_nodata_val, crop_boxes[level],
                    masked_images_vec[level], blend_weights_vec[level],
                    global_params, model_params,
                    orig_dems[level], initial_albedo,
                    lit_image_mask, curvature_in_shadow_weight,
                    // Quantities that will float
                    dems[level], albedos[level], cameras,
                    opt.image_exposures_vec,
                    opt.image_haze_vec,
                    scaled_sun_posns,
                    adjustments, opt.model_coeffs_vec);

      // TODO: Study this. Discarding the coarse DEM and exposure so
      // keeping only the cameras seem to work better.
      // Note that we overwrite dems[level-1] by resampling the coarser
      // dems[level], but we keep orig_dems[level-1] from the beginning.
      if (level > 0) {
        for (int dem_iter = 0; dem_iter < num_dems; dem_iter++) {
          if (!opt.fix_dem)
            interp_image(dems[level][dem_iter],    sub_scale, dems[level-1][dem_iter]);
          if (opt.float_albedo)
            interp_image(albedos[level][dem_iter], sub_scale, albedos[level-1][dem_iter]);
        }
      }
      
    }

  } ASP_STANDARD_CATCHES;
  
  VW_OUT(DebugMessage, "asp") << "Number of times we used the global lock: "
                              << g_num_locks << std::endl;

  sw_total.stop();
  vw_out() << "Total elapsed time: " << sw_total.elapsed_seconds() << " s." << std::endl;
 
}
