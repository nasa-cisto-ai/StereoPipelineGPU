#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

// This tool uses libpointmatcher for alignment,
// https://github.com/ethz-asl/libpointmatcher
// Copyright (c) 2010--2012,
// Francois Pomerleau and Stephane Magnenat, ASL, ETHZ, Switzerland
// You can contact the authors at <f dot pomerleau at gmail dot com> and
// <stephane at magnenat dot net>
// This tool also uses the Fast Global Registration software, under the MIT license
// https://github.com/IntelVCL/FastGlobalRegistration

// All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL ETH-ASL BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <ceres/ceres.h>
#include <ceres/loss_function.h>

// Can't do much about external warnings except hide them
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations"
#include <FastGlobalRegistration/app.h>
#pragma GCC diagnostic pop

#include <vw/Core/Stopwatch.h>
#include <vw/Math/EulerAngles.h>
#include <vw/FileIO/DiskImageView.h>
#include <vw/Cartography/Datum.h>
#include <vw/Cartography/GeoReference.h>
#include <vw/Cartography/PointImageManipulation.h>
#include <vw/FileIO/DiskImageUtils.h>
#include <vw/Core/CmdUtils.h>
#include <asp/Core/Common.h>
#include <asp/Core/Macros.h>
#include <asp/Core/PointUtils.h>
#include <asp/Core/InterestPointMatching.h>
#include <asp/Tools/pc_align_utils.h>

#include <limits>
#include <cstring>
#include <thread>
#include <omp.h>

#include <boost/filesystem.hpp>
#include <boost/algorithm/string.hpp>
namespace fs = boost::filesystem;
namespace po = boost::program_options;

using namespace vw;
using namespace std;
using namespace vw::cartography;
using namespace asp;

typedef double RealT; // We will use doubles in libpointmatcher.
typedef PointMatcher<RealT> PM;
typedef PM::DataPoints DP;
using namespace PointMatcherSupport;

const double BIG_NUMBER = 1e+300; // libpointmatcher does not like here the largest double

/// Options container for the pc_align tool
struct Options : public vw::GdalWriteOptions {
  // Input
  string reference, source, init_transform_file, alignment_method, config_file,
    datum, csv_format_str, csv_proj4_str, match_file, hillshade_options,
    ipfind_options, ipmatch_options, fgr_options;
  Vector2 initial_transform_ransac_params;
  PointMatcher<RealT>::Matrix init_transform;
  int    num_iter,
         max_num_reference_points,
         max_num_source_points;
  double diff_translation_err,
         diff_rotation_err,
         max_disp,
         outlier_ratio,
         semi_major_axis,
    semi_minor_axis, initial_rotation_angle;
  bool   compute_translation_only,
         dont_use_dem_distances,
         save_trans_source,
         save_trans_ref,
         highest_accuracy,
         verbose;
  std::string initial_ned_translation, hillshading_transform;
  
  // Output
  string out_prefix;

  Options() : max_disp(-1.0), verbose(true){}
  
  /// Return true if the reference file is a DEM file and this option is not disabled
  bool use_dem_distances() const { return ( (asp::get_cloud_type(this->reference) == "DEM") && !dont_use_dem_distances); }
};

void handle_arguments( int argc, char *argv[], Options& opt ) {
  po::options_description general_options("");
  general_options.add_options()
    ("initial-transform",        po::value(&opt.init_transform_file)->default_value(""),
                                 "The file containing the transform to be used as an initial guess. It can come from a previous run of the tool.")
    ("num-iterations",           po::value(&opt.num_iter)->default_value(1000),
                                 "Maximum number of iterations.")
    ("diff-rotation-error",      po::value(&opt.diff_rotation_err)->default_value(1e-8),
                                 "Change in rotation amount below which the algorithm will stop (if translation error is also below bound), in degrees.")
    ("diff-translation-error",   po::value(&opt.diff_translation_err)->default_value(1e-3),
                                 "Change in translation amount below which the algorithm will stop (if rotation error is also below bound), in meters.")
    ("max-displacement",         po::value(&opt.max_disp)->default_value(0.0),
                                 "Maximum expected displacement of source points as result of alignment, in meters (after the initial guess transform is applied to the source points). Used for removing gross outliers in the source point cloud.")
    ("outlier-ratio",            po::value(&opt.outlier_ratio)->default_value(0.75),
                                 "Fraction of source (movable) points considered inliers (after gross outliers further than max-displacement from reference points are removed).")
    ("max-num-reference-points", po::value(&opt.max_num_reference_points)->default_value(100000000),
                                 "Maximum number of (randomly picked) reference points to use.")
    ("max-num-source-points",    po::value(&opt.max_num_source_points)->default_value(100000),
                                 "Maximum number of (randomly picked) source points to use (after discarding gross outliers).")
    ("alignment-method",         po::value(&opt.alignment_method)->default_value("point-to-plane"),
                                 "The type of iterative closest point method to use. [point-to-plane, point-to-point, similarity-point-to-plane, similarity-point-to-point, fgr, least-squares, similarity-least-squares]")
    ("highest-accuracy",         po::bool_switch(&opt.highest_accuracy)->default_value(false)->implicit_value(true),
                                 "Compute with highest accuracy for point-to-plane (can be much slower).")
    ("csv-format",               po::value(&opt.csv_format_str)->default_value(""), asp::csv_opt_caption().c_str())
    ("csv-proj4",                po::value(&opt.csv_proj4_str)->default_value(""),
                                 "The PROJ.4 string to use to interpret the entries in input CSV files.")
    ("datum",                    po::value(&opt.datum)->default_value(""),
                                 "Use this datum for CSV files instead of auto-detecting it. Options: WGS_1984, D_MOON (1,737,400 meters), D_MARS (3,396,190 meters), MOLA (3,396,000 meters), NAD83, WGS72, and NAD27. Also accepted: Earth (=WGS_1984), Mars (=D_MARS), Moon (=D_MOON).")
    ("semi-major-axis",          po::value(&opt.semi_major_axis)->default_value(0),
                                 "Explicitly set the datum semi-major axis in meters.")
    ("semi-minor-axis",          po::value(&opt.semi_minor_axis)->default_value(0),
                                 "Explicitly set the datum semi-minor axis in meters.")
    ("output-prefix,o",          po::value(&opt.out_prefix)->default_value("run/run"),
                                 "Specify the output prefix.")
    ("compute-translation-only", po::bool_switch(&opt.compute_translation_only)->default_value(false)->implicit_value(true),
                                 "Compute the transform from source to reference point cloud as a translation only (no rotation).")
    ("save-transformed-source-points", po::bool_switch(&opt.save_trans_source)->default_value(false)->implicit_value(true),
                                  "Apply the obtained transform to the source points so they match the reference points and save them.")
    ("save-inv-transformed-reference-points", po::bool_switch(&opt.save_trans_ref)->default_value(false)->implicit_value(true),
     "Apply the inverse of the obtained transform to the reference points so they match the source points and save them.")

    ("initial-ned-translation", po::value(&opt.initial_ned_translation)->default_value(""),
                                 "Initialize the alignment transform based on a translation with this vector in the North-East-Down coordinate system around the centroid of the reference points. Specify it in quotes, separated by spaces or commas.")
    ("initial-rotation-angle", po::value(&opt.initial_rotation_angle)->default_value(0),
                                 "Initialize the alignment transform as the rotation with this angle (in degrees) around the axis going from the planet center to the centroid of the point cloud. If --initial-ned-translation is also specified, the translation gets applied after the rotation.")

    ("initial-transform-from-hillshading", po::value(&opt.hillshading_transform)->default_value(""), "If both input clouds are DEMs, find interest point matches among their hillshaded versions, and use them to compute an initial transform to apply to the source cloud before proceeding with alignment. Specify here the type of transform, as one of: 'similarity' (rotation + translation + scale), 'rigid' (rotation + translation) or 'translation'. See the options further down for tuning this.")
    ("hillshade-options", po::value(&opt.hillshade_options)->default_value("--azimuth 300 --elevation 20 --align-to-georef"), "Options to pass to the hillshade program when computing the transform from hillshading.")
    ("ipfind-options", po::value(&opt.ipfind_options)->default_value("--ip-per-image 1000000 --interest-operator sift --descriptor-generator sift"), "Options to pass to the ipfind program when computing the transform from hillshading.")
    ("ipmatch-options", po::value(&opt.ipmatch_options)->default_value("--inlier-threshold 100 --ransac-iterations 10000 --ransac-constraint similarity"), "Options to pass to the ipmatch program when computing the transform from hillshading.")
    ("match-file", po::value(&opt.match_file)->default_value(""), "Compute a translation + rotation + scale transform from the source to the reference point cloud using manually selected point correspondences from the reference to the source (obtained for example using stereo_gui). It may be desired to change --initial-transform-ransac-params if it rejects as outliers some manual matches.")
    ("initial-transform-ransac-params", po::value(&opt.initial_transform_ransac_params)->default_value(Vector2(10000, 1.0), "num_iter factor"),
     "When computing an initial transform based on hillshading, use "
     "this number of RANSAC iterations and outlier factor. A smaller factor "
     "will reject more outliers.")
    ("fgr-options", po::value(&opt.fgr_options)->default_value("div_factor: 1.4 use_absolute_scale: 0 max_corr_dist: 0.025 iteration_number: 100 tuple_scale: 0.95 tuple_max_cnt: 10000"), "Options to pass to the Fast Global Registration algorithm, if used.")
    
    ("no-dem-distances",         po::bool_switch(&opt.dont_use_dem_distances)->default_value(false)->implicit_value(true),
                                 "For reference point clouds that are DEMs, don't take advantage of the fact that it is possible to interpolate into this DEM when finding the closest distance to it from a point in the source cloud and hence the error metrics.")

    ("config-file",              po::value(&opt.config_file)->default_value(""),
     "This is an advanced option. Read the alignment parameters from a configuration file, in the format expected by libpointmatcher, over-riding the command-line options.");

  //("verbose", po::bool_switch(&opt.verbose)->default_value(false)->implicit_value(true),
  // "Print debug information");

  general_options.add( vw::GdalWriteOptionsDescription(opt) );

  po::options_description positional("");
  positional.add_options()
    ("reference", po::value(&opt.reference), "The reference (fixed) point cloud/DEM.")
    ("source",    po::value(&opt.source),    "The source (movable) point cloud/DEM.");

  po::positional_options_description positional_desc;
  positional_desc.add("reference", 1);
  positional_desc.add("source",    1);

  string usage("--max-displacement arg [other options] <reference cloud> <source cloud> -o <output prefix>");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line( argc, argv, opt, general_options, general_options,
                             positional, positional_desc, usage,
                             allow_unregistered, unregistered );

  if ( opt.reference.empty() || opt.source.empty() )
    vw_throw( ArgumentErr() << "Missing input files.\n" << usage << general_options );

  if ( opt.out_prefix.empty() )
    vw_throw( ArgumentErr() << "Missing output prefix.\n" << usage << general_options );

  if ( opt.max_disp == 0.0 )
    vw_throw( ArgumentErr() << "The max-displacement option was not set. "
              << "Use -1 if it is desired not to use it.\n" << usage << general_options );

  if ( opt.num_iter < 0 )
    vw_throw( ArgumentErr() << "The number of iterations must be non-negative.\n"
              << usage << general_options );

  if ( (opt.semi_major_axis != 0 && opt.semi_minor_axis == 0) ||
       (opt.semi_minor_axis != 0 && opt.semi_major_axis == 0) ){

    vw_throw( ArgumentErr() << "One of the semi-major or semi-minor axes"
              << " was specified, but not the other one.\n"
              << usage << general_options );
  }

  if (opt.semi_major_axis < 0 || opt.semi_minor_axis < 0){
    vw_throw( ArgumentErr() << "The semi-major and semi-minor axes cannot "
                            << "be negative.\n" << usage << general_options );
  }

  if (opt.datum != "" && opt.semi_major_axis != 0 && opt.semi_minor_axis != 0 ){
    vw_throw( ArgumentErr() << "Both the datum string and datum semi-axes were "
                            << "specified. At most one needs to be set.\n"
                            << usage << general_options );
  }

  if ((opt.initial_ned_translation != "" || opt.initial_rotation_angle != 0)
      && opt.init_transform_file != "")
    vw_throw( ArgumentErr()
              << "Cannot specify an initial transform both from a file "
              << "and as a NED vector or rotation angle.\n");

  if ( (opt.hillshading_transform != "" || opt.match_file != "") &&
       (opt.initial_ned_translation != "" || opt.init_transform_file != "" ||
        opt.initial_rotation_angle != 0)) {
    vw_throw( ArgumentErr() << "Cannot both specify an initial transform "
              << "and expect one to be computed automatically.\n");
  }

  // Create the output directory
  vw::create_out_dir(opt.out_prefix);

  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);

  // Read the initial transform
  opt.init_transform = PointMatcher<RealT>::Matrix::Identity(DIM + 1, DIM + 1);
  if (opt.init_transform_file != ""){
    asp::read_transform(opt.init_transform, opt.init_transform_file);
    vw_out() << std::setprecision(16) << "Initial guess transform:\n" << opt.init_transform << endl;
    vw_out() << std::setprecision(8); // undo the higher precision
  }

  if (opt.alignment_method != "point-to-plane"            &&
      opt.alignment_method != "point-to-point"            &&
      opt.alignment_method != "similarity-point-to-point" &&
      opt.alignment_method != "similarity-point-to-plane" &&
      opt.alignment_method != "fgr"                       &&
      opt.alignment_method != "least-squares"             &&
      opt.alignment_method != "similarity-least-squares"
      )
    vw_throw( ArgumentErr() << "Only the following alignment methods are supported: "
	      << "point-to-plane, point-to-point, similarity-point-to-point, "
              << "similarity-point-to-plane, fgr, least-squares, and similarity-least-squares.\n"
	      << usage << general_options );

  if (opt.alignment_method != "point-to-plane"            &&
      opt.alignment_method != "point-to-point"            &&
      opt.alignment_method != "similarity-point-to-point" &&
      opt.alignment_method != "similarity-point-to-plane" &&
      opt.compute_translation_only) {
    vw_throw( ArgumentErr() << "The option --compute-translation-only is only applicable to point-to-plane, point-to-point, similarity-point-to-point, and similarity-point-to-plane alignment.\n"
	      << usage << general_options );
  }
  
  if ( (opt.alignment_method == "least-squares" ||
	opt.alignment_method == "similarity-least-squares")
       && asp::get_cloud_type(opt.reference) != "DEM")
    vw_throw( ArgumentErr()
	      << "Least squares alignment can be used only when the "
	      << "reference cloud is a DEM.\n" );

  int num_iter  = opt.initial_transform_ransac_params[0];
  double factor = opt.initial_transform_ransac_params[1];
  if (num_iter < 1 || factor <= 0.0)
    vw_throw( ArgumentErr() << "Invalid values were provided for "
              << "--initial-transform-ransac-params.\n");
}

/// Compute output statistics for pc_align
void calc_stats(string label, PointMatcher<RealT>::Matrix const& dists){

  VW_ASSERT(dists.rows() == 1,
            LogicErr() << "Expecting only one row.");

  vector<double> errs(dists.cols()*dists.rows());
  int count = 0;
  for (int col = 0; col < dists.cols(); col++){
    //for (int row = 0; row < dists.rows(); row++){
    errs[count] = dists(0, col);
    count++;
    //}
  }
  sort(errs.begin(), errs.end());

  int len = errs.size();
  vw_out() << "Number of errors: " << len << endl;
  if (len == 0)
    return;

  double p16 = errs[std::min(len-1, (int)round(len*0.16))];
  double p50 = errs[std::min(len-1, (int)round(len*0.50))];
  double p84 = errs[std::min(len-1, (int)round(len*0.84))];
  vw_out() << label << ": error percentile of smallest errors (meters):"
           << " 16%: " << p16 << ", 50%: " << p50 << ", 84%: " << p84 << endl;

  double a25 = calc_mean(errs,   len/4), a50  = calc_mean(errs, len/2);
  double a75 = calc_mean(errs, 3*len/4), a100 = calc_mean(errs, len);
  vw_out() << label << ": mean of smallest errors (meters):"
           << " 25%: "  << a25 << ", 50%: "  << a50
           << ", 75%: " << a75 << ", 100%: " << a100 << endl;
}

/// Extracts the full GCC coordinate of a single point from a LibPointMatcher point cloud.
/// - The shift converts from the normalized coordinate to the actual GCC coordinate.
/// - No bounds checking is performed on the point index.
Vector3 get_cloud_gcc_coord(DP const& point_cloud, vw::Vector3 const& shift, int index) {
  Vector3 gcc_coord;
  for (int row = 0; row < DIM; ++row)
     gcc_coord[row] = point_cloud.features(row, index) + shift[row];
  return gcc_coord;
}

/// Write the output points as xyz values in binary, to be used by
/// https://github.com/IntelVCL/FastGlobalRegistration
void dump_bin(string const& file, DP const & data){

  vw_out() << "Writing: "   << data.features.cols()
           << " points to " << file << std::endl;

  FILE* fid = fopen(file.c_str(), "wb");
  int nV = data.features.cols(),
    nDim = 3; // tmp!
  fwrite(&nV, sizeof(int), 1, fid);
  fwrite(&nDim, sizeof(int), 1, fid);
  for (int c = 0; c < data.features.cols(); c++){
    float xyz[3];
    for (int r = 0; r < 3; r++) xyz[r] = data.features(r, c);
    fwrite(xyz, sizeof(float), 3, fid);

    // That code needs features
    fwrite(xyz, sizeof(float), 3, fid);
    
  }
  fclose(fid);
  
}

// Save a cloud to disk for debugging
void debug_save_point_cloud(DP const& point_cloud, GeoReference const& geo,
                            Vector3 const& shift,
                            string const& output_file){

  int numPts = point_cloud.features.cols();

  vw_out() << "Writing: " << numPts << " to " << output_file << endl;
  ofstream outfile( output_file.c_str() );
  outfile.precision(18);

  for (int col = 0; col < numPts; col++){
    Vector3 P = get_cloud_gcc_coord(point_cloud, shift, col);

    Vector3 llh = geo.datum().cartesian_to_geodetic(P); // lon-lat-height
    outfile << llh[1] << ',' << llh[0] << ',' << llh[2] << endl;
  }
  outfile.close();
}

/// Save the transform and its inverse.
void write_transforms(Options const& opt,
                     PointMatcher<RealT>::Matrix const& T){

  string transFile = opt.out_prefix + "-transform.txt";
  vw_out() << "Writing: " << transFile << endl;
  write_transform(T, transFile);

  string iTransFile = opt.out_prefix + "-inverse-transform.txt";
  PointMatcher<RealT>::Matrix invT = T.inverse();
  vw_out() << "Writing: " << iTransFile << endl;
  write_transform(invT, iTransFile);
}

/// Save the lon, lat, radius/height, and error. Use a format
/// consistent with the input CSV format.
void save_errors(DP const& point_cloud,
                 PointMatcher<RealT>::Matrix const& errors,
                 string const& output_file,
                 Vector3 const& shift,
                 GeoReference const& geo,
                 asp::CsvConv const& csv_conv,
                 bool is_lola_rdr_format,
                 double median_longitude){

  vw_out() << "Writing: " << output_file << std::endl;

  VW_ASSERT(point_cloud.features.cols() == errors.cols(),
            ArgumentErr() << "Expecting as many errors as source points.");

  ofstream outfile( output_file.c_str() );
  outfile.precision(16);

  // Write the header line
  if (csv_conv.is_configured()){
    outfile << "# " << csv_conv.write_header_string(",") << "error (meters)" << endl;
  }else{
    if (is_lola_rdr_format)
      outfile << "# longitude,latitude,radius (km),error (meters)" << endl;
    else
      outfile << "# latitude,longitude,height above datum (meters),error (meters)" << endl;
  }

  // Save the datum, may be useful to know what it was
  if (geo.datum().name() != UNSPECIFIED_DATUM){
    outfile << "# " << geo.datum() << std::endl;
    outfile << "# Projection: " << geo.overall_proj4_str() << std::endl;
  }

  int numPts = point_cloud.features.cols();
  for (int col = 0; col < numPts; col++){
    Vector3 P = get_cloud_gcc_coord(point_cloud, shift, col);

    if (csv_conv.is_configured()){
      Vector3 csv = csv_conv.cartesian_to_csv(P, geo, median_longitude);
      outfile << csv[0] << ',' << csv[1] << ',' << csv[2]
              << "," << errors(0, col) << endl;
    }else{
      Vector3 llh = geo.datum().cartesian_to_geodetic(P); // lon-lat-height
      llh[0] += 360.0*round((median_longitude - llh[0])/360.0); // 360 deg adjustment

      if (is_lola_rdr_format)
        outfile << llh[0] << ',' << llh[1] << ',' << norm_2(P)/1000.0
                << "," << errors(0, col) << endl;
      else
        outfile << llh[1] << ',' << llh[0] << ',' << llh[2]
                << "," << errors(0, col) << endl;
    }
  }
  outfile.close();
}

/// Like PM::ICP::filterGrossOutliersAndCalcErrors, except comparing to a DEM instead.
/// - The point cloud is in GCC coordinates with point_cloud_shift subtracted from each point.
/// - The output is put in the "errors" vector for each point.
/// - If there is a problem computing the point error, a very large number is used as a flag.
void calcErrorsWithDem(DP          const& point_cloud,
                       vw::Vector3 const& point_cloud_shift,
                       vw::cartography::GeoReference        const& georef,
                       vw::ImageViewRef< PixelMask<float> > const& dem,
                       std::vector<double> &errors) {

  // Initialize output error storage
  const std::int64_t num_pts = point_cloud.features.cols();
  errors.resize(num_pts);

  // Loop through every point in the point cloud
  double dem_height_here;
  for (std::int64_t i = 0; i < num_pts; i++){
    // Extract and un-shift the point to get the real GCC coordinate
    Vector3 gcc_coord = get_cloud_gcc_coord(point_cloud, point_cloud_shift, i);

    // Convert from GDC to GCC
    Vector3 llh = georef.datum().cartesian_to_geodetic(gcc_coord); // lon-lat-height

    // Interpolate the point at this location
    if (!interp_dem_height(dem, georef, llh, dem_height_here)) {
      // If we did not intersect the DEM, record a flag error value here.
      errors[i] = BIG_NUMBER;
    }
    else { // Success, the error is the absolute height difference
      errors[i] = std::abs(llh[2] - dem_height_here);
    }

  } // End loop through all points

}

template<class F>
void extract_rotation_translation(F       * transform, 
				  Quat    & rotation,
				  Vector3 & translation){

  
  Vector3 axis_angle;
  for (int i = 0; i < 3; i++){
    translation[i] = transform[i];
    axis_angle[i]  = transform[i+3];
  }
  rotation = axis_angle_to_quaternion(axis_angle);
}

// Convert a point clould to the format expected by FGR
void export_to_fgr(DP const & data, fgr::Points& pts, fgr::Feature & feat){

  pts.clear();
  feat.clear();
  for (int c = 0; c < data.features.cols(); c++){

    Eigen::Vector3f pts_v;
    for (int r = 0; r < 3; r++) pts_v[r] = data.features(r, c);

    pts.push_back(pts_v);

    // fgr expects features in addition to points. This works well enough,
    // but need to get to the bottom of whether they are necessary.
    feat.push_back(pts_v); 
  }
  
}

// Parse a string like:
// div_factor: 1.4 use_absolute_scale: 0 max_corr_dist: 0.025 iteration_number: 100 tuple_scale: 0.95 tuple_max_cnt: 10000
void parse_fgr_options(std::string const & options,
                       double            & div_factor,
                       bool              & use_absolute_scale,
                       double            & max_corr_dist,
                       int               & iteration_number,
                       float             & tuple_scale,
                       int               & tuple_max_cnt){

  // Initialize the outputs
  div_factor         = -1;
  use_absolute_scale = false;
  max_corr_dist      = -1;
  iteration_number   = -1;
  tuple_scale        = -1;
  tuple_max_cnt      = -1;

  std::istringstream is(options);
  std::string name, val;
  while( is >> name >> val){
    if (name.find("div_factor") != std::string::npos)
      div_factor = atof(val.c_str());
    if (name.find("use_absolute_scale") != std::string::npos)
      use_absolute_scale = atof(val.c_str());
    if (name.find("max_corr_dist") != std::string::npos)
      max_corr_dist = atof(val.c_str());
    if (name.find("iteration_number") != std::string::npos)
      iteration_number = atof(val.c_str());
    if (name.find("tuple_scale") != std::string::npos)
      tuple_scale = atof(val.c_str());
    if (name.find("tuple_max_cnt") != std::string::npos)
      tuple_max_cnt = atof(val.c_str());
  }
  
  // Sanity check
  if (div_factor <= 0 || max_corr_dist < 0 || iteration_number < 0 || tuple_scale <= 0 ||
      tuple_max_cnt <= 0) {
    vw_throw( ArgumentErr() << "Could not parse correctly --fgr-options.");
  }
}
  
/// Compute alignment using FGR
PointMatcher<RealT>::Matrix
fgr_alignment(DP const & source_point_cloud, DP const & ref_point_cloud, Options const& opt) {

  // Parse the options and initialize the FGR object
  double  div_factor; 
  bool    use_absolute_scale;
  double  max_corr_dist;
  int     iteration_number;
  float   tuple_scale;
  int     tuple_max_cnt;
  parse_fgr_options(opt.fgr_options,  
                    div_factor, use_absolute_scale, max_corr_dist, iteration_number,  
                    tuple_scale, tuple_max_cnt);
  fgr::CApp app(div_factor, use_absolute_scale, max_corr_dist, iteration_number,  
                tuple_scale, tuple_max_cnt);

  // Intermediate data
  fgr::Points pts;
  fgr::Feature feat;

  // Pass the reference cloud to FGR
  export_to_fgr(ref_point_cloud, pts, feat);
  app.LoadFeature(pts, feat);

  // Pass the source cloud to FGR
  export_to_fgr(source_point_cloud, pts, feat);
  app.LoadFeature(pts, feat);

  // Perform alignment
  app.NormalizePoints();
  app.AdvancedMatching();
  app.OptimizePairwise(true);
  Eigen::Matrix4f S = app.GetOutputTrans();

  // Export the transform
  PointMatcher<RealT>::Matrix T = PointMatcher<RealT>::Matrix::Identity(DIM + 1, DIM + 1);
  if (T.cols() != S.cols() || T.rows() != S.rows()) 
    vw_throw( LogicErr() << "Error: size mis-match in FGR.\n");
  for (int row = 0; row < T.rows(); row++) {
    for (int col = 0; col < T.cols(); col++) {
      T(row, col) = S(row, col);
    }
  }

  return T;
}
  
// Discrepancy between a 3D point with the rotation to be solved
// applied to it, and its projection straight down onto the DEM. Used
// with the least squares method of finding the best transform between
// clouds.
struct PointToDemError {
  PointToDemError(Vector3 const& point,
		 ImageViewRef< PixelMask<float> > const& dem,
		 cartography::GeoReference const& geo):
    m_point(point), m_dem(dem), m_geo(geo){}

  template <typename F>
  bool operator()(const F* const transform, const F* const scale, F* residuals) const {

    // Default residuals are zero, if we can't project into the DEM
    residuals[0] = F(0.0);

    // Extract the translation, and rotation
    Vector3 translation;
    Quat rotation;
    extract_rotation_translation(transform, rotation, translation);

    Vector3 trans_point = scale[0]*rotation.rotate(m_point) + translation;
    
    // Convert from GDC to GCC
    Vector3 llh = m_geo.datum().cartesian_to_geodetic(trans_point); // lon-lat-height

    // Interpolate the point at this location
    double dem_height_here;
    if (!interp_dem_height(m_dem, m_geo, llh, dem_height_here)) {
      // If we did not intersect the DEM, record a flag error value here.
      residuals[0] = F(0.0);
      return true;
    }
    
    residuals[0] = llh[2] - dem_height_here;
    return true;
  }
  
  // Factory to hide the construction of the CostFunction object from
  // the client code.
  static ceres::CostFunction* Create(Vector3 const& point,
				     ImageViewRef< PixelMask<float> > const& dem,
				     vw::cartography::GeoReference const& geo){
    return (new ceres::NumericDiffCostFunction<PointToDemError,
	    ceres::CENTRAL, 1, 6, 1>
	    (new PointToDemError(point, dem, geo)));
  }

  Vector3                                  m_point;
  ImageViewRef< PixelMask<float> > const & m_dem;    // alias
  cartography::GeoReference        const & m_geo;    // alias
};

/// Compute alignment using least squares
PointMatcher<RealT>::Matrix
least_squares_alignment(DP const& source_point_cloud, // Should not be modified
			vw::Vector3 const& point_cloud_shift,
			vw::cartography::GeoReference        const& dem_georef,
			vw::ImageViewRef< PixelMask<float> > const& dem_ref,
			Options const& opt) {

  ceres::Problem problem;

  // The final transform as a axis angle and translation pair
  std::vector<double> transform(6, 0.0);

  double scale = 1.0;
  
  // Add a residual block for every source point
  const std::int64_t num_pts = source_point_cloud.features.cols();

  // Loop through every point in the point cloud
  for (std::int64_t i = 0; i < num_pts; i++){
    
    // Extract and un-shift the point to get the real GCC coordinate
    Vector3 gcc_coord = get_cloud_gcc_coord(source_point_cloud, point_cloud_shift, i);

    ceres::CostFunction* cost_function =
      PointToDemError::Create(gcc_coord, dem_ref, dem_georef);
    ceres::LossFunction* loss_function = new ceres::CauchyLoss(0.5); // NULL;
    problem.AddResidualBlock(cost_function, loss_function, &transform[0], &scale);
    
  } // End loop through all points

  if (opt.alignment_method == "least-squares") {
    // Only solve for rotation and translation
    problem.SetParameterBlockConstant(&scale);
  }
  
  ceres::Solver::Options options;
  options.gradient_tolerance = 1e-16;
  options.function_tolerance = 1e-16;
  options.max_num_iterations = opt.num_iter;
  options.minimizer_progress_to_stdout = 1;
  options.num_threads = opt.num_threads;
  options.linear_solver_type = ceres::SPARSE_SCHUR;

  // Solve the problem
  ceres::Solver::Summary summary;
  ceres::Solve(options, &problem, &summary);

  vw_out() << summary.FullReport() << "\n" << std::endl;

  Quat rotation;
  Vector3 translation;
  extract_rotation_translation(&transform[0], rotation, translation);
  vw::Matrix<double,3,3> rot_matrix = rotation.rotation_matrix();
  
  PointMatcher<RealT>::Matrix T = PointMatcher<RealT>::Matrix::Identity(DIM + 1, DIM + 1);
  for (int row = 0; row < DIM; row++){
    for (int col = 0; col < DIM; col++){
      T(row, col) = scale*rot_matrix(col, row);
    }
  }

  for (int row = 0; row < DIM; row++)
    T(row, DIM) = translation[row];

  // This transform is in the world coordinate system (as that's the natural
  // coord system for the DEM). Transform it to the internal shifted coordinate
  // system.
  T = apply_shift(T, point_cloud_shift);

  return T;
}

/// Filters out all points from point_cloud with an error entry higher than cutoff
void filterPointsByError(DP & point_cloud, PointMatcher<RealT>::Matrix &errors, double cutoff) {

  DP input_copy = point_cloud; // Make a copy of the input DP object

  // Init LPM data structure
  const int input_point_count = point_cloud.features.cols();
  if (errors.cols() != input_point_count)
    vw_throw( LogicErr() << "Error: error size does not match point count size!\n");
  point_cloud.features.conservativeResize(DIM+1, input_point_count);
  point_cloud.featureLabels = form_labels<double>(DIM);

  // Loop through all the input points and copy them to the output if they pass the test
  std::int64_t points_count = 0;
  for (std::int64_t col = 0; col < input_point_count; ++col) {

    if (errors(0,col) > cutoff) {
      //vw_out() << "Throwing out point " << col << " for having error " << errors(0,col) << "\n";
      continue; // Error too high, don't add this point
    }

    // Copy this point to the output LPM structure
    for (std::int64_t row = 0; row < DIM; row++)
      point_cloud.features(row, points_count) = input_copy.features(row, col);
    point_cloud.features(DIM, points_count) = 1; // Extend to be a homogenous coordinate
    ++points_count; // Update output point count

  } // End loop through points

  // Finalize the LPM data structure
  point_cloud.features.conservativeResize(Eigen::NoChange, points_count);

}

// Note: The LPM matrix type used to store errors only ever has a single row.

/// Updates an LPM error matrix to use the DEM-based error for each point if it is lower.
void update_best_error(std::vector<double>         const& dem_errors,
                       PointMatcher<RealT>::Matrix      & lpm_errors) {
  std::int64_t num_points = lpm_errors.cols();
  if (dem_errors.size() != static_cast<size_t>(num_points))
    vw_throw( LogicErr() << "Error: error size does not match point count size!\n");
  //vw_out() << "Updating error...\n";

  // Loop through points
  for (std::int64_t col = 0; col < num_points; col++){
    // Use the DEM error if it is less
    if (dem_errors[col] < lpm_errors(0,col)) {
      //vw_out() << "DEM error = " << dem_errors[col] << ", LPM error = " << lpm_errors(0,col) << std::endl;
      lpm_errors(0, col) = dem_errors[col];
    }
  }

}


/// Compute the distance from source_point_cloud to the reference points.
double compute_registration_error(DP          const& ref_point_cloud,
                                  DP               & source_point_cloud, // Should not be modified
                                  PM::ICP          & pm_icp_object, // Must already be initialized
                                  vw::Vector3 const& shift,
                                  vw::cartography::GeoReference        const& dem_georef,
                                  vw::ImageViewRef< PixelMask<float> > const& dem_ref,
                                  Options const& opt,
                                  PointMatcher<RealT>::Matrix &error_matrix) {
  Stopwatch sw;
  sw.start();

  // Always start by computing the error using LPM
  // Use a big number to make sure no points are filtered!
  pm_icp_object.filterGrossOutliersAndCalcErrors(ref_point_cloud, BIG_NUMBER,
                                                 source_point_cloud, error_matrix);

  if (opt.use_dem_distances()) {
    // Compute the distance from each point to the DEM
    std::vector<double> dem_errors;
    calcErrorsWithDem(source_point_cloud, shift, dem_georef, dem_ref, dem_errors);

    // For each point use the lower of the two calculated errors.
    update_best_error(dem_errors, error_matrix);

    // We compute the error in two passes for two reasons:
    // 1 - Get the most accurate distance for each point in all cases.
    // 2 - Help fill in distances where the DEM has holes.
  }

  sw.stop();
  return sw.elapsed_seconds();
}

/// Points in source_point_cloud farther than opt.max_disp from the reference cloud are deleted.
void filter_source_cloud(DP          const& ref_point_cloud,
                         DP               & source_point_cloud,
                         PM::ICP          & pm_icp_object, // Must already be initialized
                         vw::Vector3 const& shift,
                         vw::cartography::GeoReference        const& dem_georef,
                         vw::ImageViewRef< PixelMask<float> > const& dem_ref,
                         Options const& opt) {

  // Filter gross outliers
  Stopwatch sw;
  sw.start();

  if (opt.verbose)
    vw_out() << "Filtering gross outliers" << endl;

  PointMatcher<RealT>::Matrix error_matrix;
  try {
    if (opt.use_dem_distances()) {
      // Compute the registration error using the best available means
      compute_registration_error(ref_point_cloud, source_point_cloud, pm_icp_object, shift,
                                 dem_georef, dem_ref, opt, error_matrix);

      filterPointsByError(source_point_cloud, error_matrix, opt.max_disp);
    } else { // LPM only method
        // Points in source_point_cloud further than opt.max_disp from ref_point_cloud are deleted!
        pm_icp_object.filterGrossOutliersAndCalcErrors(ref_point_cloud, opt.max_disp,
                                                       source_point_cloud, error_matrix);
    }
  }catch(const PointMatcher<RealT>::ConvergenceError & e){
    vw_throw( ArgumentErr() << "Error: No points left in source cloud after filtering. Consider increasing --max-displacement and/or see the documentation.\n");
  }


  sw.stop();
  if (opt.verbose)
    vw_out() << "Filtering gross outliers took " << sw.elapsed_seconds() << " [s]" << endl;
}


Eigen::Matrix3d vw_matrix3_to_eigen(vw::Matrix3x3 const& vw_matrix) {
  Eigen::Matrix3d out;
  out(0,0) = vw_matrix(0,0);
  out(0,1) = vw_matrix(0,1);
  out(0,2) = vw_matrix(0,2);
  out(1,0) = vw_matrix(1,0);
  out(1,1) = vw_matrix(1,1);
  out(1,2) = vw_matrix(1,2);
  out(2,0) = vw_matrix(2,0);
  out(2,1) = vw_matrix(2,1);
  out(2,2) = vw_matrix(2,2);
  return out;
}

Eigen::Vector3d vw_vector3_to_eigen(vw::Vector3 const& vw_vector) {
  return Eigen::Vector3d(vw_vector[0], vw_vector[1], vw_vector[2]);
}

// Need this to placate libpointmatcher.
std::string alignment_method_fallback(std::string const& alignment_method){
  if (alignment_method == "least-squares" || alignment_method == "similarity-least-squares" ||
      alignment_method == "fgr") 
    return "point-to-plane";
  return alignment_method;
}

// Hillshade the reference and source DEMs, and use them to find
// interest point matches among the hillshaded images.  These will be
// used later to find a rotation + translation + scale transform.
std::string find_matches_from_hillshading(Options & opt, std::string const& curr_exec_path){

  // First, this works only for DEMs
  if (asp::get_cloud_type(opt.reference) != "DEM" ||
      asp::get_cloud_type(opt.source) != "DEM" )
    vw_throw( ArgumentErr() << "Cannot find an initial transform using hillshading "
              << "unless both point clouds are DEMs. Use point2dem to first create "
              << "DEMs from the input point clouds. Then this transform can be used "
              << "with the original clouds.\n" );

  // Find the needed executables
  std::string hillshade_path = vw::program_path("hillshade", curr_exec_path);
  std::string ipfind_path    = vw::program_path("ipfind",    curr_exec_path);
  std::string ipmatch_path   = vw::program_path("ipmatch",   curr_exec_path);

  // Hillshade the reference
  std::string ref_hillshade = opt.out_prefix + "-reference_hillshade.tif";
  std::string cmd = hillshade_path + " " + opt.hillshade_options + " "
    + opt.reference + " -o " + ref_hillshade;
  vw_out() << cmd << std::endl;
  std::string ans = vw::exec_cmd(cmd.c_str());
  vw_out() << ans << std::endl;
  
  // Hillshade the source
  std::string source_hillshade = opt.out_prefix + "-source_hillshade.tif";
  cmd = hillshade_path + " " + opt.hillshade_options + " "
    + opt.source + " -o " + source_hillshade;
  vw_out() << cmd << std::endl;
  ans = vw::exec_cmd(cmd.c_str());
  vw_out() << ans << std::endl;

  // IP find
  cmd = ipfind_path + " " + opt.ipfind_options + " " + ref_hillshade + " " + source_hillshade;
  vw_out() << cmd << std::endl;
  ans = vw::exec_cmd(cmd.c_str());
  vw_out() << ans << std::endl;

  // IP match
  std::string ref_ip    = fs::path(ref_hillshade).replace_extension(".vwip").string();
  std::string source_ip = fs::path(source_hillshade).replace_extension(".vwip").string();

  cmd = ipmatch_path + " " + opt.ipmatch_options + " "
    + ref_hillshade + " " + ref_ip + " " + source_hillshade + " " + source_ip + " -o "
    + opt.out_prefix;
  vw_out() << cmd << std::endl;
  ans = vw::exec_cmd(cmd.c_str());
  vw_out() << ans << std::endl;

  // The name of the file where the matches are written to
  std::string match_file = vw::ip::match_filename(opt.out_prefix, ref_hillshade, source_hillshade);
  
  return match_file;
}

// Compute an initial source to reference transform based on tie points (interest point matches).
PointMatcher<RealT>::Matrix
initial_transform_from_match_file(std::string const& ref_file,
                                  std::string const& source_file,
                                  std::string const& match_file,
                                  std::string const& hillshading_transform,
                                  Vector2 initial_transform_ransac_params){
  
  if (asp::get_cloud_type(ref_file) != "DEM" ||
      asp::get_cloud_type(source_file) != "DEM" )
    vw_throw( ArgumentErr() << "The alignment transform computation based on manually chosen point matches only works for DEMs. Use point2dem to first create DEMs from the input point clouds.\n" );

  vector<vw::ip::InterestPoint> ref_ip, source_ip;
  vw_out() << "Reading match file: " << match_file << "\n";
  vw::ip::read_binary_match_file(match_file, ref_ip, source_ip);

  DiskImageView<float> ref(ref_file);
  vw::cartography::GeoReference ref_geo;
  bool has_ref_geo = vw::cartography::read_georeference(ref_geo, ref_file);
  double ref_nodata = -std::numeric_limits<double>::max();
  vw::read_nodata_val(ref_file, ref_nodata);

  DiskImageView<float> source(source_file);
  vw::cartography::GeoReference source_geo;
  bool has_source_geo = vw::cartography::read_georeference(source_geo, source_file);
  double source_nodata = -std::numeric_limits<double>::max();
  vw::read_nodata_val(source_file, source_nodata);

  if (!has_ref_geo || !has_source_geo)
    vw_throw( ArgumentErr() << "One of the inputs is not a valid DEM.\n" );

  if (DIM != 3)
    vw_throw( ArgumentErr() << "Expecting DIM = 3.\n" );

  // Go from pixels to 3D points
  int num_matches = ref_ip.size();
  vw::Matrix<double> points_ref(DIM, num_matches), points_src(DIM, num_matches);
  typedef vw::math::MatrixCol<vw::Matrix<double>> ColView;
  int count = 0;
  for (int match_id = 0; match_id < num_matches; match_id++) {
    int ref_x = ref_ip[match_id].x;
    int ref_y = ref_ip[match_id].y;
    if (ref_x < 0 || ref_x >= ref.cols()) continue;
    if (ref_y < 0 || ref_y >= ref.rows()) continue;
    double ref_h = ref(ref_x, ref_y);
    // Check for no-data and NaN pixels
    if (ref_h <= ref_nodata || ref_h != ref_h) continue;

    int source_x = source_ip[match_id].x;
    int source_y = source_ip[match_id].y;
    if (source_x < 0 || source_x >= source.cols()) continue;
    if (source_y < 0 || source_y >= source.rows()) continue;
    double source_h = source(source_x, source_y);
    // Check for no-data and NaN pixels
    if (source_h <= source_nodata || source_h != source_h) continue;

    Vector2 ref_ll  = ref_geo.pixel_to_lonlat(Vector2(ref_x, ref_y));
    Vector3 ref_xyz = ref_geo.datum()
      .geodetic_to_cartesian(Vector3(ref_ll[0], ref_ll[1], ref_h));

    Vector2 source_ll = source_geo.pixel_to_lonlat(Vector2(source_x, source_y));
    Vector3 source_xyz = source_geo.datum()
      .geodetic_to_cartesian(Vector3(source_ll[0], source_ll[1], source_h));

    // Store in matrices
    ColView col_ref(points_ref, count); 
    ColView col_src(points_src, count);
    col_ref = ref_xyz;
    col_src = source_xyz;
    
    count++;
  }

  if (count < 3)
    vw_throw( ArgumentErr() << "Not enough valid matches were found.\n");

  // Resize the matrix to keep only the valid points. Find the transform.
  points_src.set_size(DIM, count, true);
  points_ref.set_size(DIM, count, true);
  vw::Matrix3x3 rotation;
  vw::Vector3   translation;
  double        scale;
  bool filter_outliers = true;
  vw::math::find_3D_transform(points_src, points_ref,
                              rotation, translation, scale,
                              hillshading_transform,
                              filter_outliers,
                              initial_transform_ransac_params);

  // Convert to pc_align transform format.
  PointMatcher<RealT>::Matrix globalT = Eigen::MatrixXd::Identity(DIM+1, DIM+1);
  globalT.block(0, 0, DIM, DIM) = vw_matrix3_to_eigen(rotation*scale);
  globalT.block(0, DIM, DIM, 1) = vw_vector3_to_eigen(translation);

  vw_out() << "Transform computed from source to reference using a match file:\n"
           << globalT << std::endl;

  return globalT;
}

void apply_transform_to_cloud(PointMatcher<RealT>::Matrix const& T, DP & point_cloud){
  for (int col = 0; col < point_cloud.features.cols(); col++) {
    point_cloud.features.col(col) = T*point_cloud.features.col(col);
  }
}


// Convert a north-east-down vector at a given location to a vector in reference
// to the center of the Earth and create a translation matrix from that vector. 
PointMatcher<RealT>::Matrix ned_to_caresian_transform(vw::cartography::Datum const& datum,
                                                      std::string& ned_str, 
                                                      vw::Vector3 const & location){

  vw::string_replace(ned_str, ",", " "); // replace any commas
  vw::Vector3 ned = vw::str_to_vec<vw::Vector3>(ned_str);

  vw::Vector3 loc_llh = datum.cartesian_to_geodetic(location);
  vw::Matrix3x3 NedToEcef = datum.lonlat_to_ned_matrix(subvector(loc_llh, 0, 2));
  vw::Vector3 xyz_shift = NedToEcef * ned;
  
  PointMatcher<RealT>::Matrix T = PointMatcher<RealT>::Matrix::Identity(DIM + 1, DIM + 1);

  // Append the transform
  for (int row = 0; row < DIM; row++)
    T(row, DIM) = xyz_shift[row];

  return T;
}

// Estimate the centroid of the reference points
vw::Vector3 estimate_ref_cloud_centroid(vw::cartography::GeoReference const& geo,
                                        CsvConv const& csv_conv,
                                        std::string const& file_name){
  Stopwatch sw;
  sw.start();
  
  PointMatcherSupport::validateFile(file_name);
  PointMatcher<RealT>::DataPoints points;

  double median_longitude = 0.0; // to convert back from xyz to lonlat
  bool verbose = false;
  bool calc_shift = false; // won't shift the points
  vw::Vector3 shift = vw::Vector3(0, 0, 0);
  vw::BBox2 dummy_box;
  bool is_lola_rdr_format;
  // Load a sample of points, hopefully enough to estimate the centroid
  // reliably.
  int num_sample_pts = 1000000;
  load_cloud(file_name, num_sample_pts, dummy_box,
	     calc_shift, shift, geo, csv_conv, is_lola_rdr_format,
	     median_longitude, verbose, points);


  int numRefPts = points.features.cols();
  Eigen::VectorXd meanRef = points.features.rowwise().sum() / numRefPts;
  
  vw::Vector3 centroid;
  for (int it = 0; it < 3; it++)
    centroid[it] = meanRef[it];
  
  sw.stop();
  vw_out() << "Centroid estimation took " << sw.elapsed_seconds() << " [s]" << endl;

  return centroid;
}

// Intersect the reference and source boxes, making sure to handle well the potential
// 360 degree offset among the two.
void adjust_and_intersect_ref_source_boxes(BBox2 & ref_box, BBox2 & source_box,
                                           std::string const& reference,
                                           std::string const& source){

  double lon_offset = 0.0;
  
  // Compute the longitude offset
  double source_mean_lon = (source_box.min().x() + source_box.max().x())/2.0;
  double ref_mean_lon    = (ref_box.min().x()    + ref_box.max().x()   )/2.0;
  lon_offset = source_mean_lon - ref_mean_lon;
  lon_offset = 360.0*round(lon_offset/360.0);

  // Move the ref box in the domain of the source box
  ref_box += Vector2(lon_offset, 0);

  // Intersect them, as pc_align will operate on their common area
  ref_box.crop(source_box);
  source_box.crop(ref_box);

  // Move back the ref box to its domain
  ref_box -= Vector2(lon_offset, 0);
  
  // Extra adjustments. These are needed since pixel_to_lonlat and
  // cartesian_to_geodetic can disagree by 360 degress. Adjust ref
  // to source and vice-versa.
  adjust_lonlat_bbox(reference, ref_box);
  adjust_lonlat_bbox(source, source_box);
}

int main( int argc, char *argv[] ) {

  // Mandatory line for Eigen
  Eigen::initParallel();

  Options opt;
  try {
    handle_arguments(argc, argv, opt);

    // Set the number of threads for OpenMP. 
    int processor_count = std::thread::hardware_concurrency();
    omp_set_dynamic(0);
    omp_set_num_threads(processor_count);
    
    // Parse the csv format string and csv projection string
    asp::CsvConv csv_conv;
    csv_conv.parse_csv_format(opt.csv_format_str, opt.csv_proj4_str);

    // Try to read the georeference/datum info
    GeoReference geo;
    std::vector<std::string> clouds;
    clouds.push_back(opt.reference);
    clouds.push_back(opt.source);
    read_georef(clouds, opt.datum, opt.csv_proj4_str,  
                opt.semi_major_axis, opt.semi_minor_axis,  
                opt.csv_format_str,  csv_conv, geo);

    // Use hillshading to create a match file
    if (opt.hillshading_transform != "" && opt.match_file == "")
      opt.match_file = find_matches_from_hillshading(opt, argv[0]);
    
    // Create a transform based on a match file, either automatically generated, or
    // user-made (normally with stereo_gui).
    if (opt.match_file != "") {
      if (opt.hillshading_transform == "") 
        opt.hillshading_transform = "similarity";
      opt.init_transform = initial_transform_from_match_file(opt.reference, opt.source,
                                                             opt.match_file,
                                                             opt.hillshading_transform,
                                                             opt.initial_transform_ransac_params);
    }

    // See if to apply an initial north-east-down translation relative
    // to the point cloud centroid, and/or a rotation around the axis
    // going from the planet center to the centroid. The rotation
    // happens first.
    if (opt.initial_rotation_angle != 0 || opt.initial_ned_translation != "") {

      vw::Vector3 centroid = estimate_ref_cloud_centroid(geo, csv_conv, opt.reference);

      // Ignore any other initializations so far
      opt.init_transform = PointMatcher<RealT>::Matrix::Identity(DIM + 1, DIM + 1);

      // Form the rotation around the axis
      Eigen::Vector3d axis(centroid[0], centroid[1], centroid[2]);
      axis.normalize();
      Eigen::Matrix3d rot
        = Eigen::AngleAxisd(opt.initial_rotation_angle * M_PI/180.0, axis).matrix();
      opt.init_transform.block(0, 0, DIM, DIM) = rot;

      // The NED translation
      if (opt.initial_ned_translation != "") 
        opt.init_transform = 
          ned_to_caresian_transform(geo.datum(), opt.initial_ned_translation, centroid)
          * opt.init_transform;
    }

    // We will use ref_box to bound the source points, and vice-versa.
    // Decide how many samples to pick to estimate these boxes.
    Stopwatch sw0;
    sw0.start();
    int num_sample_pts = std::max(4000000,
                                  std::max(opt.max_num_source_points,
                                           opt.max_num_reference_points)/4);
    num_sample_pts = std::min(9000000, num_sample_pts); // avoid being slow
    
    // Compute GDC bounding box of the source and reference clouds.
    vw_out() << "Computing the intersection of the bounding boxes "
             << "of the reference and source points using " 
             << num_sample_pts << " sample points.\n";
    BBox2 ref_box, source_box, trans_ref_box, trans_source_box;

    PointMatcher<RealT>::Matrix inv_init_trans = opt.init_transform.inverse();
    calc_extended_lonlat_bbox(geo, num_sample_pts, csv_conv,
                              opt.reference, opt.max_disp, inv_init_trans,
                              ref_box, trans_ref_box);
    calc_extended_lonlat_bbox(geo, num_sample_pts, csv_conv,
                              opt.source, opt.max_disp, opt.init_transform,
                              source_box, trans_source_box);

    // When boxes are huge, it is hard to do the optimization of intersecting
    // them, as they may differ not by 0 or 360, but by 180. Better do nothing
    // in that case. The solution may degrade a bit, as we may load points
    // not in the intersection of the boxes, but at least it won't be wrong.
    // In this case, there is a chance the boxes were computed wrong anyway.
    if (ref_box.width() > 180.0 || source_box.width() > 180.0) {
      vw_out() << "Warning: Your input point clouds are spread over more than half the planet. "
               << "It is suggested that they be cropped, to get more accurate results. "
               << "Giving up on estimating their bounding boxes and filtering outliers "
               << "based on them.\n";
      ref_box = BBox2();
      source_box = BBox2();
    }
    
    vw_out() << "Reference box: " << ref_box << std::endl;
    vw_out() << "Source box:    " << source_box << std::endl;
    
    if (!ref_box.empty() && !source_box.empty()) {
      adjust_and_intersect_ref_source_boxes(ref_box, trans_source_box, opt.reference, opt.source);
      adjust_and_intersect_ref_source_boxes(trans_ref_box, source_box, opt.reference, opt.source);
    }
    
    vw_out() << "Intersection reference box:  " << ref_box    << std::endl;
    vw_out() << "Intersection source    box:  " << source_box << std::endl;
    
    sw0.stop();
    vw_out() << "Intersection of bounding boxes took " << sw0.elapsed_seconds() << " [s]" << endl;

    // Load the point clouds. We will shift both point clouds by the
    // centroid of the first one to bring them closer to origin.

    // Load the subsampled reference point cloud.
    Vector3 shift;
    bool   calc_shift = true; // Shift points so the first point is (0,0,0)
    bool   is_lola_rdr_format = false;   // may get overwritten
    double mean_ref_longitude    = 0.0;  // may get overwritten
    double mean_source_longitude = 0.0;  // may get overwritten
    Stopwatch sw1;
    sw1.start();
    DP ref_point_cloud;
    load_cloud(opt.reference, opt.max_num_reference_points, ref_box,
               calc_shift, shift, geo, csv_conv, is_lola_rdr_format,
               mean_ref_longitude, opt.verbose, ref_point_cloud);
    sw1.stop();
    if (opt.verbose)
      vw_out() << "Loading the reference point cloud took "
               << sw1.elapsed_seconds() << " [s]" << endl;
    //ref_point_cloud.save(outputBaseFile + "_ref.vtk");

    // Load the subsampled source point cloud. If the user wants
    // to filter gross outliers in the source points based on
    // max_disp, load a lot more points than asked, filter based on
    // max_disp, then resample to the number desired by the user.
    int num_source_pts = opt.max_num_source_points;
    if (opt.max_disp > 0.0)
      num_source_pts = max(num_source_pts, 50000000);
    calc_shift = false; // Use the same shift used for the reference point cloud
    Stopwatch sw2;
    sw2.start();
    DP source_point_cloud;
    load_cloud(opt.source, num_source_pts, source_box, 
	      calc_shift, shift, geo, csv_conv, is_lola_rdr_format,
	      mean_source_longitude, opt.verbose, source_point_cloud);
    sw2.stop();
    if (opt.verbose)
      vw_out() << "Loading the source point cloud took "
               << sw2.elapsed_seconds() << " [s]" << endl;

    // So far we shifted by first point in reference point cloud to reduce
    // the magnitude of all loaded points. Now that we have loaded all
    // points, shift one more time, to place the centroid of the
    // reference at the origin.
    // Note: If this code is ever converting to using floats,
    // the operation below needs to be re-implemented to be accurate.
    int numRefPts = ref_point_cloud.features.cols();
    Eigen::VectorXd meanRef = ref_point_cloud.features.rowwise().sum() / numRefPts;
    ref_point_cloud.features.topRows(DIM).colwise()    -= meanRef.head(DIM);
    source_point_cloud.features.topRows(DIM).colwise() -= meanRef.head(DIM);
    for (int row = 0; row < DIM; row++)
      shift[row] += meanRef(row); // Update the shift variable as well as the points
    if (opt.verbose)
      vw_out() << "Data shifted internally by subtracting: " << shift << std::endl;

    // The point clouds are shifted, so shift the initial transform as well.
    PointMatcher<RealT>::Matrix initT = apply_shift(opt.init_transform, shift);

    // If the reference point cloud came from a DEM, also load the data in DEM format.
    cartography::GeoReference dem_georef;
    vw::ImageViewRef< PixelMask<float> > reference_dem_ref;
    if (opt.use_dem_distances()) {
      vw_out() << "Loading reference as DEM." << endl;
      // Load the dem, then wrap it inside an ImageViewRef object.
      // - This is done because the actual DEM type cannot be created without being initialized.
      InterpolationReadyDem reference_dem(load_interpolation_ready_dem(opt.reference, dem_georef));
      reference_dem_ref.reset(reference_dem);
    }

    // Now all of the input data is loaded.

    // Filter the reference and initialize the reference tree
    double elapsed_time;
    PM::ICP icp; // LibpointMatcher object

    Stopwatch sw3;
    if (opt.verbose)
      vw_out() << "Building the reference cloud tree." << endl;
    sw3.start();
    icp.initRefTree(ref_point_cloud, alignment_method_fallback(opt.alignment_method),
		    opt.highest_accuracy, false /*opt.verbose*/);
    sw3.stop();
    if (opt.verbose)
      vw_out() << "Reference point cloud processing took " << sw3.elapsed_seconds() << " [s]\n";

    // Apply the initial guess transform to the source point cloud.
    apply_transform_to_cloud(initT, source_point_cloud);
    
    PointMatcher<RealT>::Matrix beg_errors;
    if (opt.max_disp > 0.0){
      // Filter gross outliers
      filter_source_cloud(ref_point_cloud, source_point_cloud, icp,
                          shift, dem_georef, reference_dem_ref, opt);
    }
    
    random_pc_subsample(opt.max_num_source_points, source_point_cloud.features);
    vw_out() << "Reducing number of source points to "
             << source_point_cloud.features.cols() << endl;

    // Write the point cloud to disk for debugging
    //debug_save_point_cloud(ref_point_cloud, geo, shift, "ref.csv");
    //dump_bin("ref.bin", ref_point_cloud);

    // Make the libpointmatcher error message clearer
    std::string libpointmatcher_error = "no point to minimize";
    std::string pc_align_error = std::string
      ("This likely means that the clouds are too far. Consider increasing the "
       "--max-displacement value to something somewhat larger than the expected "
       "length of the displacement that may be needed to align the clouds.\n");
    
    try {
      elapsed_time = compute_registration_error(ref_point_cloud, source_point_cloud, icp,
                                                shift, dem_georef, reference_dem_ref,
                                                opt, beg_errors);
    } catch(std::exception const& e) {
      std::string error = e.what();
      if (error.find(libpointmatcher_error) != std::string::npos)
        error += ".\n" + pc_align_error; // clarify the error
      vw_throw(ArgumentErr() << error);
    }
    
    calc_stats("Input", beg_errors);
    if (opt.verbose)
      vw_out() << "Initial error computation took " << elapsed_time << " [s]" << endl;

    // Compute the transformation to align the source to reference.
    Stopwatch sw4;
    sw4.start();
    PointMatcher<RealT>::Matrix Id = PointMatcher<RealT>::Matrix::Identity(DIM + 1, DIM + 1);
    if (opt.config_file == ""){
      // Read the options from the command line
      icp.setParams(opt.out_prefix, opt.num_iter, opt.outlier_ratio,
                    (2.0*M_PI/360.0)*opt.diff_rotation_err, // convert to radians
                    opt.diff_translation_err, alignment_method_fallback(opt.alignment_method),
                    false/*opt.verbose*/);
    }else{
      vw_out() << "Will read the options from: " << opt.config_file << endl;
      ifstream ifs(opt.config_file.c_str());
      if (!ifs.good())
        vw_throw( ArgumentErr() << "Cannot open configuration file: "
                  << opt.config_file << "\n" );
      icp.loadFromYaml(ifs);
    }

    // We bypass calling ICP if the user explicitely asks for 0 iterations.
    PointMatcher<RealT>::Matrix T = Id;
    if (opt.num_iter > 0){
      if (opt.alignment_method == "fgr") {
        T = fgr_alignment(source_point_cloud, ref_point_cloud, opt);
      } else if (opt.alignment_method == "point-to-plane" ||
                 opt.alignment_method == "point-to-point" ||
                 opt.alignment_method == "similarity-point-to-point" ||
                 opt.alignment_method == "similarity-point-to-plane") {
        // Use libpointmatcher
        try {
          T = icp(source_point_cloud, ref_point_cloud, Id, opt.compute_translation_only);
        } catch(std::exception const& e) {
          std::string error = e.what();
          if (error.find(libpointmatcher_error) != std::string::npos)
            error += ".\n" + pc_align_error; // clarify the error
          vw_throw(ArgumentErr() << error);
        }
        
        vw_out() << "Match ratio: "
		 << icp.errorMinimizer->getWeightedPointUsedRatio() << endl;
      }else if (opt.alignment_method == "least-squares" ||
                opt.alignment_method == "similarity-least-squares"){
        /// Compute alignment using least squares
	T = least_squares_alignment(source_point_cloud, shift,
				    dem_georef, reference_dem_ref, opt);
      }else
        vw_throw( ArgumentErr() << "Unknown alignment method: " << opt.alignment_method);
    }
    sw4.stop();
    if (opt.verbose)
      vw_out() << "Alignment took " << sw4.elapsed_seconds() << " [s]" << endl;

    // Transform the source to make it close to reference.
    DP trans_source_point_cloud(source_point_cloud);
    apply_transform_to_cloud(T, trans_source_point_cloud);

    // Calculate by how much points move as result of T
    double max_obtained_disp = calc_max_displacment(source_point_cloud, trans_source_point_cloud);
    Vector3 source_ctr_vec, source_ctr_llh;
    Vector3 trans_xyz, trans_ned, trans_llh;
    vw::Matrix3x3 NedToEcef;
    calc_translation_vec(initT, source_point_cloud, trans_source_point_cloud, shift,
			 geo.datum(), source_ctr_vec, source_ctr_llh,
                         trans_xyz, trans_ned, trans_llh, NedToEcef);

    // For each point, compute the distance to the nearest reference point.
    PointMatcher<RealT>::Matrix end_errors;
    elapsed_time = compute_registration_error(ref_point_cloud, trans_source_point_cloud, icp,
                                              shift, dem_georef, reference_dem_ref, opt,
					      end_errors);
    calc_stats("Output", end_errors);
    if (opt.verbose)
      vw_out() << "Final error computation took " << elapsed_time << " [s]" << endl;

    // We must apply to T the initial guess transform
    PointMatcher<RealT>::Matrix combinedT = T*initT;

    // Go back to the original coordinate system, undoing the shift
    PointMatcher<RealT>::Matrix globalT = apply_shift(combinedT, -shift);

    // Print statistics
    vw_out() << std::setprecision(16)
             << "Alignment transform (origin is planet center):" << endl << globalT << endl;
    vw_out() << std::setprecision(8); // undo the higher precision

    vw_out() << "Centroid of source points (Cartesian, meters): " << source_ctr_vec << std::endl;
    // Swap lat and lon, as we want to print lat first
    std::swap(source_ctr_llh[0], source_ctr_llh[1]);
    vw_out() << "Centroid of source points (lat,lon,z): " << source_ctr_llh << std::endl;
    vw_out() << std::endl;

    vw_out() << "Translation vector (Cartesian, meters): " << trans_xyz << std::endl;
    vw_out() << "Translation vector (North-East-Down, meters): "
             << trans_ned << std::endl;
    vw_out() << "Translation vector magnitude (meters): " << norm_2(trans_xyz)
             << std::endl;
    vw::vw_out() << "Maximum displacement of points between the source "
                 << "cloud with any initial transform applied to it and the "
                 << "source cloud after alignment to the reference: " 
                 << max_obtained_disp << " m" << std::endl;
    if (opt.max_disp > 0 && opt.max_disp < max_obtained_disp) {
      vw_out() << "Warning: The input --max-displacement value is smaller than the "
               << "final observed displacement. It may be advised to increase the former "
               << "and rerun the tool.\n";
    }

    // Swap lat and lon, as we want to print lat first
    std::swap(trans_llh[0], trans_llh[1]);
    vw_out() << "Translation vector (lat,lon,z): " << trans_llh << std::endl;
    vw_out() << std::endl;

    Matrix3x3 rot;
    for (int r = 0; r < DIM; r++)
      for (int c = 0; c < DIM; c++)
        rot(r, c) = globalT(r, c);

    double scale = pow(det(rot), 1.0/3.0);
    for (int r = 0; r < DIM; r++)
      for (int c = 0; c < DIM; c++)
        rot(r, c) /= scale;

    // Subtract one before printing the scale, to see a lot of digits of precision
    vw_out() << "Transform scale - 1 = " << (scale-1.0) << std::endl;
    
    Matrix3x3 rot_NED = inverse(NedToEcef) * rot * NedToEcef;
   
    Vector3 euler_angles = math::rotation_matrix_to_euler_xyz(rot) * 180/M_PI;
    Vector3 euler_angles_NED = math::rotation_matrix_to_euler_xyz(rot_NED) * 180/M_PI;
    Vector3 axis_angles = math::matrix_to_axis_angle(rot) * 180/M_PI;
    vw_out() << "Euler angles (degrees): " << euler_angles  << endl;
    vw_out() << "Euler angles (North-East-Down, degrees): " << euler_angles_NED  << endl;
    vw_out() << "Axis of rotation and angle (degrees): "
             << axis_angles/norm_2(axis_angles) << ' '
             << norm_2(axis_angles) << endl;

    
    Stopwatch sw5;
    sw5.start();
    write_transforms(opt, globalT);

    if (opt.save_trans_ref){
      string trans_ref_prefix = opt.out_prefix + "-trans_reference";
      save_trans_point_cloud(opt, opt.reference, trans_ref_prefix,
                             geo, csv_conv, globalT.inverse());
    }

    if (opt.save_trans_source){
      string trans_source_prefix = opt.out_prefix + "-trans_source";
      save_trans_point_cloud(opt, opt.source, trans_source_prefix,
                             geo, csv_conv, globalT);
    }

    save_errors(source_point_cloud, beg_errors,  opt.out_prefix + "-beg_errors.csv",
                shift, geo, csv_conv, is_lola_rdr_format, mean_source_longitude);
    save_errors(trans_source_point_cloud, end_errors,  opt.out_prefix + "-end_errors.csv",
                shift, geo, csv_conv, is_lola_rdr_format, mean_source_longitude);

    if (opt.verbose) vw_out() << "Writing: " << opt.out_prefix
      + "-iterationInfo.csv" << std::endl;

    sw5.stop();
    if (opt.verbose) vw_out() << "Saving to disk took "
                              << sw5.elapsed_seconds() << " [s]" << endl;

  } ASP_STANDARD_CATCHES;

  return 0;
}
