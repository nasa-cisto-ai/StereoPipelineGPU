#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file bundle_adjust.cc

// TODO(oalexan1): Break this up into several files grouped by functionality.
// Also for bundle_adjust.h. See existing BundleAdjustCamera.cc and
// BundleAdjustUtils.cc.
#include <vw/Camera/CameraUtilities.h>
#include <vw/Core/CmdUtils.h>
#include <vw/FileIO/MatrixIO.h>
#include <asp/Core/Macros.h>
#include <asp/Sessions/StereoSession.h>
#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/Sessions/CameraUtils.h>
#include <asp/Core/StereoSettings.h>
#include <asp/Core/PointUtils.h>
#include <asp/Core/IpMatchingAlgs.h> // Lightweight header for ip matching
#include <asp/Tools/bundle_adjust.h>
#include <asp/Camera/CsmModel.h>
#include <asp/Core/OutlierProcessing.h>
#include <asp/Core/DataLoader.h>

#include <vw/InterestPoint/Matcher.h>

#include <xercesc/util/PlatformUtils.hpp>

namespace po = boost::program_options;
namespace fs = boost::filesystem;

using namespace vw;
using namespace vw::camera;
using namespace vw::ba;

typedef boost::shared_ptr<asp::StereoSession> SessionPtr;
typedef CameraRelationNetwork<JFeature> CRNJ;

/// Write a csm camera state file to disk.
void write_csm_output_file(Options const& opt, int icam,
                           std::string const& adjustFile, 
                           asp::BAParams const& param_storage) {
  
  CameraAdjustment cam_adjust(param_storage.get_camera_ptr(icam));
  
  AdjustedCameraModel adj_cam(vw::camera::unadjusted_model(opt.camera_models[icam]),
                              cam_adjust.position(), cam_adjust.pose());
  
  vw::Matrix4x4 ecef_transform = adj_cam.ecef_transform();
  std::string csmFile          = asp::csmStateFile(adjustFile);
  asp::CsmModel * csm_model    = asp::csm_model(opt.camera_models[icam], opt.stereo_session);
  csm_model->saveTransformedState(csmFile, ecef_transform);
}

// Save pinhole camera positions and orientations in a single file.
// This is useful if there are thousands of cameras.
// TODO(oalexan1): Add here initial and final prefix.
// TODO(oalexan1): This must work only for pinhole.
void saveCameraReport(Options const& opt, asp::BAParams const& param_storage,
                      vw::cartography::Datum const& datum, 
                      std::string const& prefix) {

  std::string output_path = opt.out_prefix + "-" + prefix + "-cameras.csv";

  vw_out() << "Writing: " << output_path << std::endl;
  std::ofstream fh(output_path.c_str());
  fh.precision(17);
  fh << "# input_cam_file, cam_ctr_x, cam_ctr_y, cam_ctr_z (ecef meters), "
     << "cam2ned rotation rows\n";
  
  int num_cameras = opt.image_files.size();

  // TODO(oalexan1): Create here a report file. Write camera name,
  // camera center, ecef position, ecef quaternion, and ned roll-pitch-yaw.
  // Use same Euler angles as in numpy. Likely eigen can do it.
  for (int icam = 0; icam < num_cameras; icam++) {

    vw::Vector3 cam_ctr;
    vw::Matrix3x3 cam2ecef;
    switch(opt.camera_type) {
    case BaCameraType_Pinhole:
      {
        // Get the camera model from the original one with parameters in
        // param_storage applied to it (which could be original ones or optimized). 
        // Note that we do not modify the original camera.
        vw::camera::PinholeModel const* in_cam
          = dynamic_cast<vw::camera::PinholeModel const*>(opt.camera_models[icam].get());
        if (in_cam == NULL)
          vw_throw(ArgumentErr() << "Expecting a pinhole camera.\n");
        // Apply current intrinsics and extrinsics to the camera
        vw::camera::PinholeModel out_cam = transformedPinholeCamera(icam, param_storage, *in_cam);
        cam_ctr = out_cam.camera_center(vw::Vector2());
        cam2ecef = out_cam.get_rotation_matrix();
        break;
      }
    case BaCameraType_OpticalBar:
      vw::vw_throw(vw::ArgumentErr() << "Saving a camera report is not implemented "
                   << "for optical bar cameras.\n");
      break;
    default:
      {
        // Apply extrinsics adjustments to a pinhole camera
        // TODO(oalexan1): Make this into a function called adjustedPinholeCamera().
        // Use it where needed.
        CameraAdjustment adjustment(param_storage.get_camera_ptr(icam));
        PinholeModel* in_cam = dynamic_cast<PinholeModel*>(opt.camera_models[icam].get());
        if (in_cam == NULL)
          vw_throw(ArgumentErr() << "Expecting a pinhole camera.\n");
        
        // Make a copy of the camera, and apply the adjustments to the copy. Need to go back
        // to the original camera to get the adjustments needed to apply.
        // TODO(oalexan1): This is a little awkward.
        PinholeModel out_cam = *in_cam;
        AdjustedCameraModel adj_cam(vw::camera::unadjusted_model(opt.camera_models[icam]),
                                    adjustment.position(), adjustment.pose());
        vw::Matrix4x4 ecef_transform = adj_cam.ecef_transform();
        out_cam.apply_transform(ecef_transform);
        cam_ctr = out_cam.camera_center(vw::Vector2());
        cam2ecef = out_cam.get_rotation_matrix();
      }
    }

    fh << opt.camera_files[icam] << ", "
       << cam_ctr[0] << ", " << cam_ctr[1] << ", " << cam_ctr[2];

    // Find the matrix for converting NED to ECEF
    vw::Vector3 loc_llh = datum.cartesian_to_geodetic(cam_ctr);
    vw::Matrix3x3 ned2ecef = datum.lonlat_to_ned_matrix(subvector(loc_llh, 0, 2));

    // How a camera moves relative to the world is given by the camera-to-world
    // matrix. That is a little counter-intuitive.
    vw::Matrix3x3 cam2ned = inverse(ned2ecef) * cam2ecef;
    for (int row = 0; row < cam2ned.rows(); row++) {
      for (int col = 0; col < cam2ned.cols(); col++) {
        fh << ", " << cam2ned(row, col);
      } 
    }
    fh << "\n";
    // See
    // https://stackoverflow.com/questions/27508242/roll-pitch-and-yaw-from-rotation-matrix-with-eigen-library
    // for how to create roll-pitch-yaw. May need to first convert the
    // camera from NED to maybe East-South-Down, which may be
    // convenient if the camera flies North-to-South and the image
    // rows go West to East. In Eigen, the roll-pitch-yaw then could
    // be found as: m.eulerAngles(2,1,0)
  }

  fh.close();

  return;
}

// Write the results to disk.
void saveResults(Options const& opt, asp::BAParams const& param_storage) {
  int num_cameras = opt.image_files.size();

  for (int icam = 0; icam < num_cameras; icam++){

    switch(opt.camera_type) {
    case BaCameraType_Pinhole:
      write_pinhole_output_file(opt, icam, param_storage);
      break;
    case BaCameraType_OpticalBar:
      write_optical_bar_output_file(opt, icam, param_storage);
      break;
    default:
      std::string adjust_file = asp::bundle_adjust_file_name(opt.out_prefix,
                                                             opt.image_files[icam],
                                                             opt.camera_files[icam]);
      vw_out() << "Writing: " << adjust_file << std::endl;
      
      CameraAdjustment cam_adjust(param_storage.get_camera_ptr(icam));
      asp::write_adjustments(adjust_file, cam_adjust.position(), cam_adjust.pose());

      // For CSM camera models export, in addition, the JSON state
      // with the adjustment applied to it.
      if (opt.stereo_session == "csm" || opt.stereo_session == "pleiades" ||
          (opt.stereo_session == "dg" && asp::stereo_settings().dg_use_csm))
        write_csm_output_file(opt, icam, adjust_file, param_storage);
    }
    
  } // End loop through cameras
  
}

// A callback to invoke at each iteration if desiring to save the cameras
// at that time.
class BaCallback: public ceres::IterationCallback {
public:
  
  BaCallback(Options const& opt, asp::BAParams const& param_storage):
    m_opt(opt), m_param_storage(param_storage){}

  virtual ceres::CallbackReturnType operator() (const ceres::IterationSummary& summary) {
    saveResults(m_opt, m_param_storage);
    return ceres::SOLVER_CONTINUE;
  }
  
private:
  Options const& m_opt;
  asp::BAParams const& m_param_storage;
};

/// Add error source for projecting a 3D point into the camera.
void add_reprojection_residual_block(Vector2 const& observation, Vector2 const& pixel_sigma,
                                     int point_index, int camera_index, 
                                     asp::BAParams & param_storage,
                                     Options const& opt,
                                     ceres::Problem & problem){

  ceres::LossFunction* loss_function;
  loss_function = get_loss_function(opt);

  boost::shared_ptr<CameraModel> camera_model = opt.camera_models[camera_index];

  double* camera = param_storage.get_camera_ptr(camera_index);
  double* point  = param_storage.get_point_ptr (point_index );

  if (opt.camera_type == BaCameraType_Other) {
    // The generic camera case
    boost::shared_ptr<CeresBundleModelBase> wrapper(new AdjustedCameraBundleModel(camera_model));
      ceres::CostFunction* cost_function =
        BaReprojectionError::Create(observation, pixel_sigma, wrapper);
      problem.AddResidualBlock(cost_function, loss_function, point, camera);

  } else { // Pinhole and optical bar

    double* center     = param_storage.get_intrinsic_center_ptr    (camera_index);
    double* focus      = param_storage.get_intrinsic_focus_ptr     (camera_index);
    double* distortion = param_storage.get_intrinsic_distortion_ptr(camera_index);

    boost::shared_ptr<CeresBundleModelBase> wrapper;

    if (opt.camera_type == BaCameraType_Pinhole) {

      boost::shared_ptr<PinholeModel> pinhole_model = 
        boost::dynamic_pointer_cast<PinholeModel>(camera_model);
      if (pinhole_model.get() == 0)
        vw::vw_throw(vw::ArgumentErr() << "Tried to add pinhole block with non-pinhole camera.");
      wrapper.reset(new PinholeBundleModel(pinhole_model));

    } else { // Optical bar

      boost::shared_ptr<vw::camera::OpticalBarModel> bar_model = 
        boost::dynamic_pointer_cast<vw::camera::OpticalBarModel>(camera_model);
      if (bar_model.get() == 0)
        vw::vw_throw( vw::ArgumentErr() << "Tried to add optical bar block with "
                      << "non-optical bar camera.");
      wrapper.reset(new OpticalBarBundleModel(bar_model));
    }

    ceres::CostFunction* cost_function =
      BaReprojectionError::Create(observation, pixel_sigma, wrapper);
    problem.AddResidualBlock(cost_function, loss_function, point, camera, 
                            center, focus, distortion);

    // Apply the residual limits
    size_t num_limits = opt.intrinsics_limits.size() / 2;
    if ((num_limits > 0) && (num_limits > wrapper->num_intrinsic_params())) {
      vw::vw_throw( vw::ArgumentErr() << "Error: Too many intrinsic limits provided!"
        << " This model has " << wrapper->num_intrinsic_params() << " intrinsic parameters.");
    }
    size_t intrin_index = 0;
    if (num_limits > 0) { // Do focus first.
      problem.SetParameterLowerBound(focus, 0, opt.intrinsics_limits[0]);
      problem.SetParameterUpperBound(focus, 0, opt.intrinsics_limits[1]);
      ++intrin_index;
    }
    while ((intrin_index < 3) && (intrin_index < num_limits)) { // Next is the two center params
      problem.SetParameterLowerBound(center, intrin_index-1,
                                     opt.intrinsics_limits[2*intrin_index    ]);
      problem.SetParameterUpperBound(center, intrin_index-1,
                                     opt.intrinsics_limits[2*intrin_index + 1]);
      ++intrin_index;
    }
    while (intrin_index < num_limits) { // Finish with the intrinsic params
      problem.SetParameterLowerBound(distortion, intrin_index-3,
                                     opt.intrinsics_limits[2*intrin_index    ]);
      problem.SetParameterUpperBound(distortion, intrin_index-3,
                                     opt.intrinsics_limits[2*intrin_index + 1]);
      ++intrin_index;
    }

    // If we don't want to solve for something, just tell Ceres not to adjust the values.
    if (opt.intrinisc_options.center_constant)
      problem.SetParameterBlockConstant(center);
    if (opt.intrinisc_options.focus_constant)
      problem.SetParameterBlockConstant(focus);
    if (opt.intrinisc_options.distortion_constant)
      problem.SetParameterBlockConstant(distortion);
  } // End non-generic camera case.

  // Fix this camera if requested
  if (opt.fixed_cameras_indices.find(camera_index) != opt.fixed_cameras_indices.end()) 
    problem.SetParameterBlockConstant(param_storage.get_camera_ptr(camera_index));
}

/// Add residual block for the error using reference xyz.
void add_disparity_residual_block(Vector3 const& reference_xyz,
                                  ImageViewRef<DispPixelT> const& interp_disp, 
                                  int left_cam_index, int right_cam_index,
                                  asp::BAParams & param_storage,
                                  Options const& opt,
                                  ceres::Problem & problem){

  ceres::LossFunction* loss_function = get_loss_function(opt);

  boost::shared_ptr<CameraModel> left_camera_model  = opt.camera_models[left_cam_index ];
  boost::shared_ptr<CameraModel> right_camera_model = opt.camera_models[right_cam_index];

  const bool inline_adjustments = (opt.camera_type != BaCameraType_Other);

  // Get the list of residual pointers that will be passed to ceres.
  std::vector<double*> residual_ptrs;
  BaDispXyzError::get_residual_pointers(param_storage,
                                        left_cam_index, right_cam_index,
                                        inline_adjustments, opt.intrinisc_options,
                                        residual_ptrs);
 if (opt.camera_type == BaCameraType_Other) {

    boost::shared_ptr<CeresBundleModelBase> left_wrapper (new AdjustedCameraBundleModel(left_camera_model ));
    boost::shared_ptr<CeresBundleModelBase> right_wrapper(new AdjustedCameraBundleModel(right_camera_model));
    ceres::CostFunction* cost_function =
      BaDispXyzError::Create(reference_xyz, interp_disp, left_wrapper, right_wrapper,
                             inline_adjustments, opt.intrinisc_options);

    problem.AddResidualBlock(cost_function, loss_function, residual_ptrs);

  } else { // Pinhole or optical bar

    boost::shared_ptr<CeresBundleModelBase> left_wrapper, right_wrapper;

    if (opt.camera_type == BaCameraType_Pinhole) {
      boost::shared_ptr<PinholeModel> left_pinhole_model = 
        boost::dynamic_pointer_cast<vw::camera::PinholeModel>(left_camera_model);
      boost::shared_ptr<PinholeModel> right_pinhole_model = 
        boost::dynamic_pointer_cast<vw::camera::PinholeModel>(right_camera_model);

      left_wrapper.reset (new PinholeBundleModel(left_pinhole_model ));
      right_wrapper.reset(new PinholeBundleModel(right_pinhole_model));

    } else { // Optical bar
      boost::shared_ptr<vw::camera::OpticalBarModel> left_bar_model = 
        boost::dynamic_pointer_cast<vw::camera::OpticalBarModel>(left_camera_model);
      boost::shared_ptr<vw::camera::OpticalBarModel> right_bar_model = 
        boost::dynamic_pointer_cast<vw::camera::OpticalBarModel>(right_camera_model);

      left_wrapper.reset (new OpticalBarBundleModel(left_bar_model ));
      right_wrapper.reset(new OpticalBarBundleModel(right_bar_model));
    }

    ceres::CostFunction* cost_function =
      BaDispXyzError::Create(reference_xyz, interp_disp, left_wrapper, right_wrapper,
                             inline_adjustments, opt.intrinisc_options);
    problem.AddResidualBlock(cost_function, loss_function, residual_ptrs);

  }
  
} // End function add_disparity_residual_block


//----------------------------------------------------------------
// Residuals functions

/// Compute the residuals
void compute_residuals(bool apply_loss_function,
                       Options const& opt,
                       asp::BAParams const& param_storage,
                       std::vector<size_t> const& cam_residual_counts,
                       size_t num_gcp_or_dem_residuals,
                       size_t num_tri_residuals,
                       std::vector<vw::Vector3> const& reference_vec,
                       ceres::Problem & problem,
                       // Output
                       std::vector<double> & residuals) {
  // TODO: Associate residuals with cameras!
  // Generate some additional diagnostic info
  double cost = 0.0;
  ceres::Problem::EvaluateOptions eval_options;
  eval_options.apply_loss_function = apply_loss_function;
  if (opt.single_threaded_cameras)
    eval_options.num_threads = 1; // ISIS must be single threaded!
  else
    eval_options.num_threads = opt.num_threads;

  problem.Evaluate(eval_options, &cost, &residuals, 0, 0);
  const size_t num_residuals = residuals.size();
  
  // Verify our book-keeping is correct
  size_t num_expected_residuals
    = (num_gcp_or_dem_residuals + num_tri_residuals) * param_storage.params_per_point();
  size_t total_num_cam_params   = param_storage.num_cameras()*param_storage.params_per_camera();
  for (size_t i=0; i<param_storage.num_cameras(); i++)
    num_expected_residuals += cam_residual_counts[i]*PIXEL_SIZE;
  if (opt.camera_weight > 0)
    num_expected_residuals += total_num_cam_params;
  if (opt.rotation_weight > 0 || opt.translation_weight > 0)
    num_expected_residuals += total_num_cam_params;
  num_expected_residuals += reference_vec.size() * PIXEL_SIZE;
  
  if (num_expected_residuals != num_residuals)
    vw_throw( LogicErr() << "Expected " << num_expected_residuals
                         << " residuals but instead got " << num_residuals);
}

/// Compute residual map by averaging all the reprojection error at a given point
void compute_mean_residuals_at_xyz(CRNJ & crn,
                                  std::vector<double> const& residuals,
                                  asp::BAParams const& param_storage,
                                  // outputs
                                  std::vector<double> & mean_residuals,
                                  std::vector<int>  & num_point_observations) {

  mean_residuals.resize(param_storage.num_points());
  num_point_observations.resize(param_storage.num_points());
  
  // Observation residuals are stored at the beginning of the residual vector in the 
  //  same order they were originally added to Ceres.
  
  size_t residual_index = 0;
  // Double loop through cameras and crn entries will give us the correct order
  for ( size_t icam = 0; icam < param_storage.num_cameras(); icam++ ) {
    typedef CameraNode<JFeature>::const_iterator crn_iter;
    for (crn_iter fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++){

      // The index of the 3D point
      int ipt = (**fiter).m_point_id;

      if (param_storage.get_point_outlier(ipt))
        continue; // skip outliers

      // Get the residual error for this observation
      double errorX         = residuals[residual_index  ];
      double errorY         = residuals[residual_index+1];
      // TODO(oalexan1): Use norm_2 below rather than average. This may
      // change the regressions.
      double residual_error = (fabs(errorX) + fabs(errorY)) / 2;
      residual_index += PIXEL_SIZE;

      // Update information for this point
      num_point_observations[ipt] += 1;
      mean_residuals        [ipt] += residual_error;
    }
  } // End double loop through all the observations

  // Do the averaging
  for (size_t i = 0; i < param_storage.num_points(); i++) {
    if (param_storage.get_point_outlier(i)) {
      // Skip outliers. But initialize to something.
      mean_residuals        [i] = std::numeric_limits<double>::quiet_NaN();
      num_point_observations[i] = std::numeric_limits<int>::quiet_NaN();
      continue;
    }
    mean_residuals[i] /= static_cast<double>(num_point_observations[i]);
  }
  
} // End function compute_mean_residuals_at_xyz

/// Write out a .csv file recording the residual error at each location on the ground
void write_residual_map(std::string const& output_prefix,
                        // Mean residual of each point
                        std::vector<double> const& mean_residuals,
                        // Num non-outlier pixels per point
                        std::vector<int> const& num_point_observations, 
                        asp::BAParams const& param_storage,
                        ControlNetwork const& cnet,
                        Options const& opt) {

  std::string output_path = output_prefix + ".csv";

  if (opt.datum.name() == asp::UNSPECIFIED_DATUM) {
    vw_out(WarningMessage) << "No datum specified, can't write file: " << output_path << ". "
                           << "With Earth satellite images add the option '-t nadirpinhole' "
                           << "to use the WGS84 datum, or else specify '--datum <planet name>'.\n";
    return;
  }
  if (mean_residuals.size() != param_storage.num_points())
    vw_throw( LogicErr() << "Point count mismatch in write_residual_map().\n");

  if (cnet.size() != param_storage.num_points()) 
    vw_throw( LogicErr()
              << "The number of stored points "
              << "does not agree with number of points in cnet.\n");
  
  // Open the output file and write the header
  vw_out() << "Writing: " << output_path << std::endl;
  std::ofstream file;
  file.open(output_path.c_str());
  file.precision(17);
  file << "# lon, lat, height_above_datum, mean_residual, num_observations\n";

  // stereo_gui counts on being able to parse the datum from this file, so
  // do not modify the line below.
  file << "# " << opt.datum << std::endl;
  
  // Now write all the points to the file
  for (size_t i = 0; i < param_storage.num_points(); i++) {

    if (param_storage.get_point_outlier(i))
      continue; // skip outliers
    
      // The final GCC coordinate of this point
      const double * point = param_storage.get_point_ptr(i);
      Vector3 xyz(point[0], point[1], point[2]);

      Vector3 llh = opt.datum.cartesian_to_geodetic(xyz);

      std::string comment = "";
      if (cnet[i].type() == ControlPoint::GroundControlPoint)
        comment = " # GCP";
      else if (cnet[i].type() == ControlPoint::PointFromDem)
        comment = " # from DEM";
      
      file << llh[0] <<", "<< llh[1] <<", "<< llh[2] <<", "<< mean_residuals[i] <<", "
           << num_point_observations[i] << comment << std::endl;
  }
  file.close();

} // End function write_residual_map


/// Write log files describing all residual errors. The order of data stored
/// in residuals must mirror perfectly the way residuals were created. 
void write_residual_logs(std::string const& residual_prefix, bool apply_loss_function,
                         Options const& opt,
                         asp::BAParams const& param_storage,
                         std::vector<size_t> const& cam_residual_counts,
                         size_t num_gcp_or_dem_residuals,
                         size_t num_tri_residuals,
                         std::vector<vw::Vector3> const& reference_vec,
                         ControlNetwork const& cnet, CRNJ & crn, 
                         ceres::Problem &problem) {
  
  std::vector<double> residuals;
  compute_residuals(apply_loss_function, opt, param_storage,
                    cam_residual_counts, num_gcp_or_dem_residuals, num_tri_residuals,
                    reference_vec, problem,
                    // Output
                    residuals);
    
  const size_t num_residuals = residuals.size();

  const std::string residual_path               = residual_prefix + "_stats.txt";
  const std::string residual_raw_pixels_path    = residual_prefix + "_raw_pixels.txt";
  const std::string residual_raw_gcp_path       = residual_prefix + "_raw_gcp.txt";
  const std::string residual_raw_cams_path      = residual_prefix + "_raw_cameras.txt";
  const std::string residual_reference_xyz_path = residual_prefix + "_reference_terrain.txt";

  // Write a report on residual errors
  std::ofstream residual_file, residual_file_raw_pixels, residual_file_raw_gcp,
    residual_file_raw_cams, residual_file_reference_xyz;
  vw_out() << "Writing: " << residual_path << std::endl;
  vw_out() << "Writing: " << residual_raw_pixels_path << std::endl;
  vw_out() << "Writing: " << residual_raw_gcp_path << std::endl;
  vw_out() << "Writing: " << residual_raw_cams_path << std::endl;
  
  residual_file.open(residual_path.c_str());
  residual_file.precision(17);
  residual_file_raw_pixels.open(residual_raw_pixels_path.c_str());
  residual_file_raw_pixels.precision(17);
  residual_file_raw_cams.open(residual_raw_cams_path.c_str());
  residual_file_raw_cams.precision(17);

  if (reference_vec.size() > 0) {
    //vw_out() << "Writing: " << residual_reference_xyz_path << std::endl;
    residual_file_reference_xyz.open(residual_reference_xyz_path.c_str());
    residual_file_reference_xyz.precision(17);
  }
  
  size_t index = 0;
  // For each camera, average together all the point observation residuals
  residual_file << "Mean and median norm of residual error and point count for cameras:\n";
  for (size_t c = 0; c < param_storage.num_cameras(); c++) {
    size_t num_this_cam_residuals = cam_residual_counts[c];
    
    // Write header for the raw file
    std::string name = opt.camera_files[c];
    if (name == "")
      name = opt.image_files[c];
    
    residual_file_raw_pixels << name << ", " << num_this_cam_residuals << std::endl;

    // All residuals are for inliers, as we do not even add a residual
    // for an outlier
    
    double mean_residual = 0; // Take average of all pixel coord errors
    std::vector<double> residual_norms;
    for (size_t i = 0; i < num_this_cam_residuals; i++) {
      double ex = residuals[index];
      ++index;
      double ey = residuals[index];
      ++index;
      double residual_norm = std::sqrt(ex * ex + ey * ey);
      mean_residual += residual_norm;
      residual_norms.push_back(residual_norm);
      residual_file_raw_pixels << ex << ", " << ey << std::endl; // Write ex, ey on raw file
    }
    // Write line for the summary file
    mean_residual /= static_cast<double>(num_this_cam_residuals);
    double median_residual = std::numeric_limits<double>::quiet_NaN();
    if (residual_norms.size() > 0) {
      std::sort(residual_norms.begin(), residual_norms.end());
      median_residual = residual_norms[residual_norms.size()/2];
    }
    
    residual_file << name                   << ", "
                  << mean_residual          << ", "
                  << median_residual        << ", "
                  << num_this_cam_residuals << std::endl;
  }
  
  residual_file_raw_pixels.close();
  
  // List the GCP residuals
  if (num_gcp_or_dem_residuals > 0) {
    residual_file_raw_gcp.open(residual_raw_gcp_path.c_str());
    residual_file_raw_gcp.precision(17);
    residual_file << "GCP or DEM residual errors:\n";
    for (size_t i = 0; i < num_gcp_or_dem_residuals; i++) {
      double mean_residual = 0; // Take average of XYZ error for each point
      residual_file_raw_gcp << i;
      for (size_t j = 0; j < param_storage.params_per_point(); j++) {
        mean_residual += fabs(residuals[index]);
        residual_file_raw_gcp << ", " << residuals[index]; // Write all values in this file
        ++index;
      }
      mean_residual /= static_cast<double>(param_storage.params_per_point());
      residual_file << i << ", " << mean_residual << std::endl;
      residual_file_raw_gcp << std::endl;
    }
    residual_file_raw_gcp.close();
  }
  
  // List the camera weight residuals
  int num_passes = int(opt.camera_weight > 0) +
    int(opt.rotation_weight > 0 || opt.translation_weight > 0);
  for (int pass = 0; pass < num_passes; pass++) {
    residual_file << "Camera weight position and orientation residual errors:\n";
    const size_t part_size = param_storage.params_per_camera()/2;
    for (size_t c=0; c<param_storage.num_cameras(); ++c) {
      residual_file_raw_cams << opt.camera_files[c];
      // Separately compute the mean position and rotation error
      double mean_residual_pos = 0, mean_residual_rot = 0;
      for (size_t j = 0; j < part_size; j++) {
        mean_residual_pos += fabs(residuals[index]);
        residual_file_raw_cams << ", " << residuals[index]; // Write all values in this file
        ++index;
      }
      for (size_t j = 0; j < part_size; j++) {
        mean_residual_rot += fabs(residuals[index]);
        residual_file_raw_cams << ", " << residuals[index]; // Write all values in this file
        ++index;
      }
      mean_residual_pos /= static_cast<double>(part_size);
      mean_residual_rot /= static_cast<double>(part_size);
    
      residual_file << opt.camera_files[c] << ", " << mean_residual_pos << ", "
                    << mean_residual_rot << std::endl;
      residual_file_raw_cams << std::endl;
    }
  }
  residual_file_raw_cams.close();
  residual_file.close();

  // List residuals for matching input terrain (lidar)
  if (reference_vec.size() > 0) {
    residual_file << "reference terrain residual errors:\n";
    residual_file_reference_xyz << "# lon, lat, height_above_datum, pixel_error_norm\n";
    for (size_t i = 0; i < reference_vec.size(); i++) {

      Vector3 llh = opt.datum.cartesian_to_geodetic(reference_vec[i]);
      double err = norm_2(Vector2(residuals[index], residuals[index + 1]));

      // Divide back the residual by the multiplier weight
      if (opt.reference_terrain_weight > 0) 
        err /= opt.reference_terrain_weight;
      
      index += PIXEL_SIZE;
      residual_file_reference_xyz << llh[0] << ", " << llh[1] << ", " << llh[2] << ", "
                                  << err << "\n";
      residual_file << i << ", " << err << "\n";
      
    }
    residual_file_reference_xyz.close();
  }

  // Keep track of number of triangulation constraint residuals but don't save those
  index += asp::BAParams::PARAMS_PER_POINT * num_tri_residuals;
  
  if (index != num_residuals)
    vw_throw( LogicErr() << "Have " << num_residuals << " residuals, but iterated through "
              << index);

  // Generate the location based file
  std::string map_prefix = residual_prefix + "_pointmap";
  std::vector<double> mean_residuals;
  std::vector<int> num_point_observations;
  compute_mean_residuals_at_xyz(crn,  residuals,  param_storage,
                                mean_residuals, num_point_observations);

  write_residual_map(map_prefix, mean_residuals, num_point_observations,
                     param_storage, cnet, opt);

} // End function write_residual_logs


// End residual functions
// ----------------------------------------------------------------

// ----------------------------------------------------------------
// Start outlier functions

/// Add to the outliers based on the large residuals
int add_to_outliers(ControlNetwork & cnet,
                    CRNJ & crn,
                    asp::BAParams & param_storage,
                    Options const& opt,
                    std::vector<size_t> const& cam_residual_counts,
                    size_t num_gcp_or_dem_residuals,
                    size_t num_tri_residuals,
                    std::vector<vw::Vector3> const& reference_vec, 
                    ceres::Problem &problem) {

  vw_out() << "Removing pixel outliers in preparation for another solver attempt.\n";

  const size_t num_points  = param_storage.num_points();
  const size_t num_cameras = param_storage.num_cameras();
  
  // Compute the reprojection error. Hence we should not add the contribution
  // of the loss function.
  bool apply_loss_function = false;
  std::vector<double> residuals;
  compute_residuals(apply_loss_function,  
                    opt, param_storage,  cam_residual_counts,  
                    num_gcp_or_dem_residuals, num_tri_residuals, reference_vec, problem,
                    // output
                    residuals);

  // Compute the mean residual at each xyz, and how many times that residual is seen
  std::vector<double> mean_residuals;
  std::vector<int   > num_point_observations;
  compute_mean_residuals_at_xyz(crn,  residuals,  param_storage,
                                // outputs
                                mean_residuals, num_point_observations);

  // The number of mean residuals is the same as the number of points,
  // of which some are outliers. Hence need to collect only the
  // non-outliers so far to be able to remove new outliers.  Need to
  // follow the same logic as when residuals were formed. And also ignore GCP.
  std::vector<double> actual_residuals;
  std::set<int> was_added;
  for ( size_t icam = 0; icam < num_cameras; icam++ ) {
    typedef CameraNode<JFeature>::const_iterator crn_iter;
    for ( crn_iter fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++ ){

      // The index of the 3D point
      int ipt = (**fiter).m_point_id;

      // skip existing outliers
      if (param_storage.get_point_outlier(ipt))
        continue; 

      // Skip gcp, those are never outliers no matter what.
      if (cnet[ipt].type() == ControlPoint::GroundControlPoint)
        continue;

      // We already encountered this residual in the previous camera
      if (was_added.find(ipt) != was_added.end()) 
        continue;
      
      was_added.insert(ipt);
      actual_residuals.push_back(mean_residuals[ipt]);
      //vw_out() << "XYZ residual " << ipt << " = " << mean_residuals[ipt] << std::endl;
    }
  } // End double loop through all the observations

  double pct      = 1.0 - opt.remove_outliers_params[0]/100.0;
  double factor   = opt.remove_outliers_params[1];
  double max_pix1 = opt.remove_outliers_params[2];
  double max_pix2 = opt.remove_outliers_params[3];

  double b, e; 
  vw::math::find_outlier_brackets(actual_residuals, pct, factor, b, e);
  vw_out() << "Outlier statistics: b = " << b << ", e = " << e << ".\n";
  
  // If this is too aggressive, the user can tame it. It is
  // unreasonable to throw out pixel residuals as small as 1 or 2
  // pixels.  We will not use the b, because the residuals start at 0.
  // - "max_pix" sets the minimum error that can be thrown out.
  e = std::min(std::max(e, max_pix1), max_pix2);

  vw_out() << "Removing as outliers points with mean reprojection error > " << e << ".\n";
  
  // Add to the outliers by reprojection error. Must repeat the same logic as above.
  // TODO(oalexan1): This removes a 3D point altogether if any reprojection
  // errors for it are big. Need to only remove bad reprojection errors
  // and keep a 3D point if it is left with at least two reprojection residuals.
  int num_outliers_by_reprojection = 0, total = 0;
  for ( size_t icam = 0; icam < num_cameras; icam++ ) {
    typedef CameraNode<JFeature>::const_iterator crn_iter;
    for ( crn_iter fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++ ){

      // The index of the 3D point
      int ipt = (**fiter).m_point_id;

      total++;
      
      // skip existing outliers
      if (param_storage.get_point_outlier(ipt))
        continue; 

      // Skip gcp
      if (cnet[ipt].type() == ControlPoint::GroundControlPoint)
        continue;

      if (mean_residuals[ipt] > e) {
        param_storage.set_point_outlier(ipt, true);
        num_outliers_by_reprojection++;
      }
    }
  } // End double loop through all the observations
  vw_out() << "Removed " << num_outliers_by_reprojection << " outliers out of "
           << total << " by reprojection error. Ratio: "
           << double(num_outliers_by_reprojection) / double(total) <<".\n";
  
  // Remove outliers by elevation limit
  int num_outliers_by_elev_or_lonlat = 0;
  if (opt.elevation_limit[0] < opt.elevation_limit[1] || !opt.lon_lat_limit.empty()) {

    for (size_t ipt = 0; ipt < param_storage.num_points(); ipt++) {

      if (cnet[ipt].type() == ControlPoint::GroundControlPoint)
        continue; // don't filter out GCP
      if (param_storage.get_point_outlier(ipt))
        continue; // skip outliers
      
      // The GCC coordinate of this point
      const double * point = param_storage.get_point_ptr(ipt);
      Vector3 xyz(point[0], point[1], point[2]);
      Vector3 llh = opt.datum.cartesian_to_geodetic(xyz);
      if (opt.elevation_limit[0] < opt.elevation_limit[1] && 
          (llh[2] < opt.elevation_limit[0] ||
           llh[2] > opt.elevation_limit[1])) {
        param_storage.set_point_outlier(ipt, true);
        num_outliers_by_elev_or_lonlat++;
        continue;
      }
      
      Vector2 lon_lat = subvector(llh, 0, 2);
      if ( !opt.lon_lat_limit.empty() && !opt.lon_lat_limit.contains(lon_lat) ) {
        param_storage.set_point_outlier(ipt, true);
        num_outliers_by_elev_or_lonlat++;
        continue;
      }
      
    }
    vw_out() << "Removed " << num_outliers_by_elev_or_lonlat
             << " outliers by elevation range and/or lon-lat range.\n";
  }

  // Remove outliers based on spatial extent. Be more generous with
  // leaving data in than what the input parameters suggest, because
  // sometimes inliers in space need not be uniformly distributed.
  double pct_factor = (3.0 + opt.remove_outliers_params[0]/100.0)/4.0; // e.g., 0.9375
  double outlier_factor = 2 * opt.remove_outliers_params[1];           // e.g., 6.0.
  std::vector<double> x_vals, y_vals, z_vals;
  for (size_t ipt = 0; ipt < param_storage.num_points(); ipt++) {
    
    if (cnet[ipt].type() == ControlPoint::GroundControlPoint)
      continue; // don't filter out GCP
    if (param_storage.get_point_outlier(ipt))
      continue; // skip outliers
    
    // The GCC coordinate of this point
    const double * point = param_storage.get_point_ptr(ipt);
    x_vals.push_back(point[0]);
    y_vals.push_back(point[1]);
    z_vals.push_back(point[2]);
  }
  vw::BBox3 estim_bdbox;
  asp::estimate_inliers_bbox(pct_factor, pct_factor, pct_factor,
                             outlier_factor,
                             x_vals, y_vals, z_vals,  
                             estim_bdbox); // output
  
  int num_box_outliers = 0;
  for (size_t ipt = 0; ipt < param_storage.num_points(); ipt++) {
    
    if (cnet[ipt].type() == ControlPoint::GroundControlPoint)
      continue; // don't filter out GCP
    if (param_storage.get_point_outlier(ipt))
      continue; // skip outliers
    
    // The GCC coordinate of this point
    const double * point = param_storage.get_point_ptr(ipt);
    Vector3 xyz(point[0], point[1], point[2]);
    if (!estim_bdbox.contains(xyz)) {
      param_storage.set_point_outlier(ipt, true);
      num_box_outliers++;
    }
  }

  vw_out() << "Removed " << num_box_outliers << " " 
           << "outlier(s) based on spatial distribution of triangulated points.\n";
  
  int num_remaining_points = num_points - param_storage.get_num_outliers();

  return num_outliers_by_reprojection + num_outliers_by_elev_or_lonlat;
}

// Find the cameras with the latest adjustments. Note that we do not modify
// opt.camera_models, but make copies as needed.
void calcOptimizedCameras(Options const& opt,
                          asp::BAParams const& param_storage,
                          std::vector<vw::CamPtr> & optimized_cams) {

  optimized_cams.clear();
  
  int num_cameras = opt.image_files.size();
  for (int icam = 0; icam < num_cameras; icam++) {
    
    // TODO(oalexan1): The logic below may need to be a function and should be called
    // in a couple other places.
    switch (opt.camera_type) {
    case BaCameraType_Pinhole:
      {
        vw::camera::PinholeModel const* in_cam
          = dynamic_cast<vw::camera::PinholeModel const*>(opt.camera_models[icam].get());
        if (in_cam == NULL)
          vw_throw(ArgumentErr() << "Expecting a pinhole camera.\n");
        vw::camera::PinholeModel * out_cam = new PinholeModel();
        *out_cam = transformedPinholeCamera(icam, param_storage, *in_cam);
        optimized_cams.push_back(vw::CamPtr(out_cam));
      }
      break;
      
    case BaCameraType_OpticalBar:
      {
        vw::camera::OpticalBarModel const* in_cam
          = dynamic_cast<vw::camera::OpticalBarModel const*>(opt.camera_models[icam].get());
        if (in_cam == NULL)
          vw_throw(ArgumentErr() << "Expecting an optical bar camera.\n");
        vw::camera::OpticalBarModel * out_cam = new OpticalBarModel();
        *out_cam = transformedOpticalBarCamera(icam, param_storage, *in_cam);
        optimized_cams.push_back(vw::CamPtr(out_cam));
      }
      break;
      
    default:
      {
        CameraAdjustment cam_adjust(param_storage.get_camera_ptr(icam));
        vw::CamPtr out_cam
          (new AdjustedCameraModel(vw::camera::unadjusted_model(opt.camera_models[icam]),
                                          cam_adjust.position(), cam_adjust.pose()));
        optimized_cams.push_back(out_cam);
      }
    }
  }
}

// End outlier functions
// ----------------------------------------------------------------
// TODO(oalexan1): Use this in jitter_solve.
// TODO(oalexan1): This needs to be done before subsampling the matches
void initial_filter_by_proj_win(Options             & opt,
                                asp::BAParams      & param_storage, 
                                ControlNetwork const& cnet) {

  // Swap y. Sometimes it is convenient to specify these on input in reverse.
  if (opt.proj_win.min().y() > opt.proj_win.max().y())
    std::swap(opt.proj_win.min().y(), opt.proj_win.max().y());

  // Set the projection. The function set_proj4_projection_str() does not set the
  // datum radii, which is confusing. Use asp::set_srs_string().
  vw::cartography::GeoReference georef;
  bool have_datum = (opt.datum.name() != asp::UNSPECIFIED_DATUM);
  bool have_input_georef = false;
  asp::set_srs_string(opt.proj_str, have_datum, opt.datum,
                      have_input_georef, georef);

  int num_points  = param_storage.num_points();
  for (int i = 0; i < num_points; i++) {
      
    if (param_storage.get_point_outlier(i))
      continue;
      
    double* point = param_storage.get_point_ptr(i);
    Vector3 xyz(point[0], point[1], point[2]);
    Vector3 llh = georef.datum().cartesian_to_geodetic(xyz);
    Vector2 proj_pt = georef.lonlat_to_point(subvector(llh, 0, 2));

    if (!opt.proj_win.contains(proj_pt))
      param_storage.set_point_outlier(i, true);
  }
}

int do_ba_ceres_one_pass(Options             & opt,
                         CRNJ                & crn,
                         bool                  first_pass,
                         asp::BAParams       & param_storage, 
                         asp::BAParams const & orig_parameters,
                         bool                & convergence_reached,
                         double              & final_cost) {

  ceres::Problem problem;

  ControlNetwork & cnet = *opt.cnet;
  const int num_cameras = param_storage.num_cameras();
  const int num_points  = param_storage.num_points();

  if ((int)crn.size() != num_cameras) 
    vw_throw(ArgumentErr() << "Book-keeping error, the size of CameraRelationNetwork "
             << "must equal the number of images.\n");
 
  convergence_reached = true;

  if (opt.proj_win != BBox2(0, 0, 0, 0) && (!opt.proj_str.empty()))
    initial_filter_by_proj_win(opt, param_storage, cnet);
  
  // How many times an xyz point shows up in the problem
  std::vector<int> count_map(num_points);
  for (int i = 0; i < num_points; i++) {
    if (param_storage.get_point_outlier(i))
      count_map[i] = 0; // skip outliers
    else
      count_map[i] = cnet[i].size(); // Get number of observations of this point.
  }

  // We will optimize multipliers of the intrinsics. This way
  // each intrinsic changes by a scale specific to it.
  // Note: If an intrinsic starts as 0, it will then stay as 0. This is documented.
  // Can be both useful and confusing.

  bool have_dem = (!opt.heights_from_dem.empty() || !opt.ref_dem.empty());
  
  // Create anchor xyz with the help of a DEM in two ways.
  // TODO(oalexan1): Study how to best pass the DEM to avoid the code
  // below not being slow. It is not clear if the DEM tiles are cached
  // when passing around an ImageViewRef.
  std::vector<Vector3> dem_xyz_vec;
  vw::cartography::GeoReference dem_georef;
  ImageViewRef<PixelMask<double>> interp_dem;
  std::set<int> outliers;
  if (have_dem) {
    for (int ipt = 0; ipt < num_points; ipt++) {
      if (param_storage.get_point_outlier(ipt))
        outliers.insert(ipt);
    }
  }
  if (opt.heights_from_dem != "") {
    asp::create_interp_dem(opt.heights_from_dem, dem_georef, interp_dem);
    asp::update_point_height_from_dem(cnet, outliers, dem_georef, interp_dem,  
                                      // Output
                                      dem_xyz_vec);
  }
  if (opt.ref_dem != "") {
    asp::create_interp_dem(opt.ref_dem, dem_georef, interp_dem);
    asp::calc_avg_intersection_with_dem(cnet, crn, outliers, opt.camera_models,
                                        dem_georef, interp_dem,
                                        // Output
                                        dem_xyz_vec);
  }
  
  // Add the cost function component for difference of pixel observations
  // - Reduce error by making pixel projection consistent with observations.
  
  // Add the various cost functions the solver will optimize over.
  std::vector<size_t> cam_residual_counts(num_cameras);
  typedef CameraNode<JFeature>::iterator crn_iter;
  for (int icam = 0; icam < num_cameras; icam++) { // Camera loop
    cam_residual_counts[icam] = 0;
    for (crn_iter fiter = crn[icam].begin(); fiter != crn[icam].end(); fiter++) { // IP loop

      // The index of the 3D point this IP is for.
      int ipt = (**fiter).m_point_id;
      if (param_storage.get_point_outlier(ipt))
        continue; // skip outliers

      VW_ASSERT(int(icam) < num_cameras,
                ArgumentErr() << "Out of bounds in the number of cameras.");
      VW_ASSERT(int(ipt)  < num_points,
                ArgumentErr() << "Out of bounds in the number of points.");

      double* point = param_storage.get_point_ptr(ipt);
      if (point[0] == 0 && point[1] == 0 && point[2] == 0) {
        // Flag points in the center of the planet as outliers
        param_storage.set_point_outlier(ipt, true);
        continue;
      }
      
      // Adjust non-GCP triangulated points based on the DEM, if
      // provided (two approaches are supported).
      bool is_gcp = (cnet[ipt].type() == ControlPoint::GroundControlPoint);
      if (have_dem && !is_gcp && dem_xyz_vec.at(ipt) != Vector3(0, 0, 0)) {
        for (int p = 0; p < 3; p++) 
          point[p] = dem_xyz_vec.at(ipt)[p]; // update the tri point based on the DEM
        cnet[ipt].set_type(ControlPoint::PointFromDem); // so we can track it later
        cnet[ipt].set_position(Vector3(point[0], point[1], point[2])); // update in the cnet too
        
        if (opt.heights_from_dem != "") {
          if (opt.heights_from_dem_weight <= 0) {
            // Fix it
            problem.SetParameterBlockConstant(point);
          }else{
            // Let it float. Later a constraint will be added.
            double s = 1.0/opt.heights_from_dem_weight;
            cnet[ipt].set_sigma(Vector3(s, s, s));
          }
          
        }else  if (opt.ref_dem != "") {
          if (opt.ref_dem_weight <= 0) {
            // Fix it
            problem.SetParameterBlockConstant(point);
          }else{
            // Let it float. Later a constraint will be added.
            double s = 1.0/opt.ref_dem_weight;
            cnet[ipt].set_sigma(Vector3(s, s, s));
          }
        }
      }

      // The observed value for the projection of point with index ipt into
      // the camera with index icam.
      Vector2 observation = (**fiter).m_location;
      Vector2 pixel_sigma = (**fiter).m_scale;

      // This is a bugfix
      if (pixel_sigma != pixel_sigma) // nan check
        pixel_sigma = Vector2(1, 1);

      double p = opt.overlap_exponent;
      if (p > 0 && count_map[ipt] > 2) {
        // Give more weight to points that are seen in more images.
        // This should not be overused. 
        double delta = pow(count_map[ipt] - 1.0, p);
        pixel_sigma /= delta;
      }

      // Call function to add the appropriate Ceres residual block.
      add_reprojection_residual_block(observation, pixel_sigma, ipt, icam,
                                      param_storage, opt, problem);
      cam_residual_counts[icam] += 1; // Track the number of residual blocks for each camera
      
    } // end iterating over points
  } // end iterating over cameras

  // Add ground control points or points based on a DEM constraint
  // Error goes up as GCP's move from their input positions.
  int num_gcp = 0, num_gcp_or_dem_residuals = 0;
  for (int ipt = 0; ipt < num_points; ipt++) {
    if (cnet[ipt].type() != ControlPoint::GroundControlPoint &&
        cnet[ipt].type() != ControlPoint::PointFromDem)
      continue; // Skip non-GCP's and points which do not need special treatment

    if (param_storage.get_point_outlier(ipt))
      continue; // skip outliers
    
    if (cnet[ipt].type() == ControlPoint::GroundControlPoint)
      num_gcp++;

    Vector3 observation = cnet[ipt].position();
    Vector3 xyz_sigma   = cnet[ipt].sigma();

    ceres::CostFunction* cost_function;
    if (!opt.use_llh_error) 
      cost_function = XYZError::Create(observation, xyz_sigma);
    else{
      Vector3 llh_sigma = xyz_sigma;
      // make lat,lon into lon,lat
      std::swap(llh_sigma[0], llh_sigma[1]);
      cost_function = LLHError::Create(observation, llh_sigma, opt.datum);
    }

    // Don't use the same loss function as for pixels since that one
    // discounts outliers and the GCP's should never be discounted.
    // The user an override this for the advanced --heights_from_dem
    // and --reference-dem options.
    ceres::LossFunction* loss_function = NULL;
    if (opt.heights_from_dem != ""      &&
        opt.heights_from_dem_weight > 0 &&
        opt.heights_from_dem_robust_threshold > 0) {
      loss_function = get_loss_function(opt, opt.heights_from_dem_robust_threshold);
    }else if (opt.ref_dem != "" &&
        opt.ref_dem_weight > 0  &&
        opt.ref_dem_robust_threshold > 0) {
      loss_function = get_loss_function(opt, opt.ref_dem_robust_threshold);
    }else{
      loss_function = new ceres::TrivialLoss();
    }
    double * point  = param_storage.get_point_ptr(ipt);
    problem.AddResidualBlock(cost_function, loss_function, point);

    num_gcp_or_dem_residuals++;

    if (opt.fix_gcp_xyz) 
      problem.SetParameterBlockConstant(point);
  } // End loop through GCP's

  // Add camera constraints
  // - Error goes up as cameras move and rotate from their input positions.
  if (opt.camera_weight > 0){
    for (int icam = 0; icam < num_cameras; icam++){
      double const* orig_cam_ptr = orig_parameters.get_camera_ptr(icam);
      ceres::CostFunction* cost_function = CamError::Create(orig_cam_ptr, opt.camera_weight);

      // Don't use the same loss function as for pixels since that one discounts
      //  outliers and the cameras should never be discounted.
      // TODO(oalexan1): This will prevent convergence in some cases!
      ceres::LossFunction* loss_function = new ceres::TrivialLoss();

      double * camera  = param_storage.get_camera_ptr(icam);
      problem.AddResidualBlock(cost_function, loss_function, camera);
    } // End loop through cameras.
  }

  // Finer level control of only rotation and translation.
  // - Error goes up as cameras move and rotate from their input positions.
  // TODO(oalexan1): This will prevent convergence in some cases as there is no attenuation
  if (opt.rotation_weight > 0 || opt.translation_weight > 0){
    for (int icam = 0; icam < num_cameras; icam++){
      double const* orig_cam_ptr = orig_parameters.get_camera_ptr(icam);
      ceres::CostFunction* cost_function
        = RotTransError::Create(orig_cam_ptr, opt.rotation_weight, opt.translation_weight);
      ceres::LossFunction* loss_function = new ceres::TrivialLoss();
      double * camera  = param_storage.get_camera_ptr(icam);
      problem.AddResidualBlock(cost_function, loss_function, camera);
    }
  }

  // TODO(oalexan1): Make this into a function in a separate file,
  // as it depends on Eigen which makes compilation even slower than
  // what it already is.
  // Add a cost function meant to tie up to known disparity
  // form left to right image and known ground truth reference terrain.
  // This was only tested for local pinhole cameras.
  // Disparity must be created with stereo -e 3 with the
  // option --unalign-disparity. If there are n images,
  // there must be n-1 disparities, from each image to the next.
  // The doc has more info in the bundle_adjust chapter.
  std::vector<ImageView<DispPixelT>> disp_vec;
  std::vector<ImageViewRef<DispPixelT>> interp_disp; 
  std::vector<vw::Vector3> reference_vec;
  if (opt.reference_terrain != "") {
    // TODO: Pass these properly
    g_max_disp_error           = opt.max_disp_error;
    g_reference_terrain_weight = opt.reference_terrain_weight;
    
    // Set up a GeoReference object using the datum, it may get modified later
    vw::cartography::GeoReference geo;
    geo.set_datum(opt.datum); // We checked for a datum earlier

    // Load the reference data
    std::vector<vw::Vector3> input_reference_vec;
    asp::load_csv_or_dem(opt.csv_format_str, opt.csv_proj4_str, opt.reference_terrain,  
                         opt.max_num_reference_points,  
                         geo,       // may change
                         input_reference_vec); // output

    if (load_reference_disparities(opt.disparity_list, disp_vec, interp_disp) != num_cameras-1)
      vw_throw(ArgumentErr() << "Expecting one less disparity than there are cameras.\n");
    
    std::vector<vw::BBox2i> image_boxes;
    for (int icam = 0; icam < num_cameras; icam++){
      DiskImageView<float> img(opt.image_files[icam]);
      BBox2i bbox = vw::bounding_box(img);
      image_boxes.push_back(bbox);
    }

    vw_out() << "Setting up the error to the reference terrain.\n";
    TerminalProgressCallback tpc("", "\t--> ");
    tpc.report_progress(0);
    double inc_amount = 1.0/double(input_reference_vec.size());

    reference_vec.clear();
    for (size_t data_col = 0; data_col < input_reference_vec.size(); data_col++) {

      vw::Vector3 reference_xyz = input_reference_vec[data_col];

      // Filter by lonlat box if provided, this is very much recommended
      // to quickly discard most points in the huge reference terrain.
      // Let's hope there is no 360 degree offset when computing
      // the longitude. 
      if ( asp::stereo_settings().lon_lat_limit != BBox2(0,0,0,0) ) {
        vw::Vector3 llh = geo.datum().cartesian_to_geodetic(reference_xyz);
        vw::Vector2 ll  = subvector(llh, 0, 2);
        if (!asp::stereo_settings().lon_lat_limit.contains(ll)) {
          continue;
        }
      }

      Vector2 left_pred, right_pred;

      // Iterate over the cameras, add a residual for each point and each camera pair.
      for (int icam = 0; icam < num_cameras - 1; icam++) {

        boost::shared_ptr<CameraModel> left_camera  = opt.camera_models[icam  ];
        boost::shared_ptr<CameraModel> right_camera = opt.camera_models[icam+1];

        try {
          left_pred  = left_camera->point_to_pixel (reference_xyz);
          right_pred = right_camera->point_to_pixel(reference_xyz);
        } catch (const camera::PointToPixelErr& e) {
          continue; // Skip point if there is a projection issue.
        }

        if ( (left_pred != left_pred) || (right_pred != right_pred) )
          continue; // nan check

        if (!interp_disp[icam].pixel_in_bounds(left_pred))
          continue; // Interp check

        DispPixelT dispPix = interp_disp[icam](left_pred[0], left_pred[1]);
        if (!is_valid(dispPix))
          continue;

        // Check if the current point projects in the cameras
        if ( !image_boxes[icam  ].contains(left_pred ) || 
             !image_boxes[icam+1].contains(right_pred)   ) {
          continue;
        }

        Vector2 right_pix = left_pred + dispPix.child();
        if (!image_boxes[icam+1].contains(right_pix)) 
          continue; // Check offset location too

        if (right_pix != right_pix || norm_2(right_pix - right_pred) > opt.max_disp_error) {
          // Ignore pixels which are too far from where they should be before optimization
          continue;
        }

        reference_vec.push_back(reference_xyz); // only the used reference points are stored here

        // Call function to select the appropriate Ceres residual block to add.
        add_disparity_residual_block(reference_xyz, interp_disp[icam],
                                     icam, icam+1, // left icam and right icam
                                     param_storage, opt, problem);
      }
      tpc.report_incremental_progress(inc_amount);
    }
    
    tpc.report_finished();
    vw_out() << "Found " << reference_vec.size() << " reference points in range.\n";
  } // End of reference terrain block

  int num_tri_residuals = 0;
  if (opt.tri_weight > 0) {
    // Add triangulation weight to make each triangulated point not move too far
    for (int ipt = 0; ipt < num_points; ipt++) {
      if (cnet[ipt].type() == ControlPoint::GroundControlPoint ||
          cnet[ipt].type() == ControlPoint::PointFromDem)
        continue; // Skip GCPs and height-from-dem points which have their own constraint
      
      if (param_storage.get_point_outlier(ipt))
        continue; // skip outliers
      
      double * point = param_storage.get_point_ptr(ipt);

      // Use as constraint the initially triangulated point
      Vector3 observation(point[0], point[1], point[2]);
      double s = 1.0/opt.tri_weight;
      Vector3 xyz_sigma(s, s, s);

      ceres::CostFunction* cost_function = XYZError::Create(observation, xyz_sigma);
      ceres::LossFunction* loss_function = get_loss_function(opt, opt.tri_robust_threshold);
      problem.AddResidualBlock(cost_function, loss_function, point);

      num_tri_residuals++;
    } // End loop through xyz
  } // end adding a triangulation constraint
  
  const size_t MIN_KML_POINTS = 50;
  size_t kmlPointSkip = 30;
  // Figure out a good KML point skip amount
  if (num_points / kmlPointSkip < MIN_KML_POINTS)
    kmlPointSkip = num_points / MIN_KML_POINTS;
  if (kmlPointSkip < 1)
    kmlPointSkip = 1;
    
  if (first_pass) {
    // Save the cnet 
    if (opt.save_cnet_as_csv) {
      std::string cnet_file = opt.out_prefix + "-cnet.csv";
      vw_out() << "Writing: " << cnet_file << std::endl;
      cnet.write_in_gcp_format(cnet_file, opt.datum);
    }
    
    
    std::string point_kml_path  = opt.out_prefix + "-initial_points.kml";
    std::string residual_prefix = opt.out_prefix + "-initial_residuals";
    vw_out() << "Writing initial condition files." << std::endl;
    bool apply_loss_function = false;
    write_residual_logs(residual_prefix, apply_loss_function, opt, param_storage, 
                        cam_residual_counts, num_gcp_or_dem_residuals, num_tri_residuals,
                        reference_vec, cnet, crn, problem);
    
    param_storage.record_points_to_kml(point_kml_path, opt.datum, 
                         kmlPointSkip, "initial_points",
                        "http://maps.google.com/mapfiles/kml/shapes/placemark_circle.png");
  }

  // Solve the problem
  ceres::Solver::Options options;
  options.gradient_tolerance  = 1e-16;
  options.function_tolerance  = 1e-16;
  options.parameter_tolerance = opt.parameter_tolerance; // default is 1e-8

  options.max_num_iterations                = opt.num_iterations;
  options.max_num_consecutive_invalid_steps = std::max(5, opt.num_iterations/5); // try hard
  options.minimizer_progress_to_stdout      = true;

  if (opt.single_threaded_cameras)
    options.num_threads = 1;
  else
    options.num_threads = opt.num_threads;

  // Use a callback function at every iteration, if desired to save the intermediate results
  BaCallback callback(opt, param_storage);
  if (opt.save_intermediate_cameras) {
    options.callbacks.push_back(&callback);
    options.update_state_every_iteration = true;
  }

  // Set solver options according to the recommendations in the Ceres solving FAQs
  options.linear_solver_type = ceres::SPARSE_SCHUR;
  if (num_cameras < 100)
    options.linear_solver_type = ceres::DENSE_SCHUR;
  if (num_cameras > 3500) {
    // This is supposed to help with speed in a certain size range
    options.use_explicit_schur_complement = true; 
    options.linear_solver_type  = ceres::ITERATIVE_SCHUR;
    options.preconditioner_type = ceres::SCHUR_JACOBI;
  }
  if (num_cameras > 7000)
    options.use_explicit_schur_complement = false; // Only matters with ITERATIVE_SCHUR

  //options.ordering_type = ceres::SCHUR;
  //options.eta = 1e-3; // FLAGS_eta;
  //options->max_solver_time_in_seconds = FLAGS_max_solver_time;
  //options->use_nonmonotonic_steps = FLAGS_nonmonotonic_steps;
  //if (FLAGS_line_search) {
  //  options->minimizer_type = ceres::LINE_SEARCH;
  //}

  vw_out() << "Starting the Ceres optimizer." << std::endl;
  ceres::Solver::Summary summary;
  ceres::Solve(options, &problem, &summary);
  final_cost = summary.final_cost;
  vw_out() << summary.FullReport() << "\n";
  if (summary.termination_type == ceres::NO_CONVERGENCE){
    // Print a clarifying message, so the user does not think that the algorithm failed.
    vw_out() << "Found a valid solution, but did not reach the actual minimum." << std::endl;
    convergence_reached = false;
  }

  // Write the condition files after each pass, as we never know which pass will be the last
  // since we may stop the passes prematurely if no more outliers are present.
  vw_out() << "Writing final condition log files." << std::endl;
  std::string residual_prefix = opt.out_prefix + "-final_residuals";
  bool apply_loss_function = false;
  write_residual_logs(residual_prefix, apply_loss_function, opt, param_storage,
                      cam_residual_counts,
                      num_gcp_or_dem_residuals, num_tri_residuals,
                      reference_vec, cnet, crn, problem);
  
  std::string point_kml_path = opt.out_prefix + "-final_points.kml";
  std::string url = "http://maps.google.com/mapfiles/kml/shapes/placemark_circle_highlight.png";
  param_storage.record_points_to_kml(point_kml_path, opt.datum, kmlPointSkip, "final_points",
                                     url);
  
  // Print the stats for GCP
  // TODO(oalexan1): This should go to a file
  if (num_gcp > 0) 
    param_storage.print_gcp_stats(cnet, opt.datum);

  // Outlier filtering
  bool remove_outliers = (opt.num_ba_passes > 1);
  if (remove_outliers)
      add_to_outliers(cnet, crn,
                      param_storage,   // in-out
                      opt, cam_residual_counts, num_gcp_or_dem_residuals,
                      num_tri_residuals, reference_vec, problem);

  // Find the cameras with the latest adjustments. Note that we do not modify
  // opt.camera_models, but make copies as needed.
  std::vector<vw::CamPtr> optimized_cams;
  calcOptimizedCameras(opt, param_storage, optimized_cams);
  
  // Calculate convergence angles. Remove the outliers flagged earlier,
  // if remove_outliers is true. Compute offsets of mapprojected matches,
  // if a DEM is given. These are done together as they rely on
  // reloading interest point matches, which is expensive so the matches
  // are used for both operations.
  std::vector<vw::Vector<float, 4>> mapprojPoints; // all points, not just stats
  std::vector<asp::MatchPairStats> convAngles, mapprojOffsets;
  std::vector<std::vector<float>> mapprojOffsetsPerCam;
  vw::cartography::GeoReference mapproj_dem_georef;
  if (!opt.mapproj_dem.empty()) {
    bool is_good = vw::cartography::read_georeference(mapproj_dem_georef, opt.mapproj_dem);
    if (!is_good) 
      vw::vw_throw(vw::ArgumentErr() << "Could not read a georeference from: "
                   << opt.mapproj_dem << ".\n");
  }
  outliers.clear(); 
  for (int i = 0; i < param_storage.num_points(); i++)
    if (param_storage.get_point_outlier(i))
      outliers.insert(i); // update this based on param_storage
  asp::matchFilesProcessing(cnet,
                            asp::BaBaseOptions(opt), // note the slicing
                            optimized_cams, remove_outliers, outliers, opt.mapproj_dem,
                            convAngles, mapprojPoints, mapprojOffsets, mapprojOffsetsPerCam);

  std::string conv_angles_file = opt.out_prefix + "-convergence_angles.txt";
  asp::saveConvergenceAngles(conv_angles_file, convAngles, opt.image_files);

  if (!opt.mapproj_dem.empty()) {
    std::string mapproj_offsets_stats_file = opt.out_prefix + "-mapproj_match_offset_stats.txt";
    std::string mapproj_offsets_file = opt.out_prefix + "-mapproj_match_offsets.txt";
    asp::saveMapprojOffsets(mapproj_offsets_stats_file, mapproj_offsets_file,
                            mapproj_dem_georef,
                            mapprojPoints,
                            mapprojOffsets, 
                            mapprojOffsetsPerCam, // will change
                            opt.image_files);
  }
  
  return 0;
} // End function do_ba_ceres_one_pass

/// Use Ceres to do bundle adjustment.
void do_ba_ceres(Options & opt, std::vector<Vector3> const& estimated_camera_gcc){

  // Try to set up the control network, ie the list of point coordinates.
  // - This triangulates from the camera models to determine the initial
  //   world coordinate estimate for each matched IP.
  opt.cnet.reset(new ControlNetwork("BundleAdjust"));
  int num_gcp = 0;
  ControlNetwork & cnet = *(opt.cnet.get()); // alias
  if (!opt.apply_initial_transform_only) {
    bool triangulate_control_points = true;
    bool success = vw::ba::build_control_network(triangulate_control_points,
                                                 cnet, opt.camera_models,
                                                 opt.image_files,
                                                 opt.match_files,
                                                 opt.min_matches,
                                                 opt.min_triangulation_angle*(M_PI/180.0),
                                                 opt.forced_triangulation_distance,
                                                 opt.max_pairwise_matches);
    if (!success) {
      vw_out() << "Failed to build a control network.\n"
               << " - Consider removing all .vwip and .match files and \n"
               << "   increasing the number of interest points per tile using\n "
               << "   --ip-per-tile, or decreasing --min-matches.\n"
               << " - Check if your images are similar enough in illumination,\n"
               << "   and if they have enough overlap.\n"   
               << "Will continue if ground control points are present.\n";
    }
    vw_out() << "Loading GCP files...\n";
    num_gcp = vw::ba::add_ground_control_points(cnet, opt.gcp_files, opt.datum);
  }
  
  // If we change the cameras, we must rebuild the control network
  bool cameras_changed = false;
  
  // If camera positions were provided for local inputs, align to them.
  const bool have_est_camera_positions = (opt.camera_position_file != "");
  if ((opt.camera_type == BaCameraType_Pinhole) && have_est_camera_positions) {
    asp::init_pinhole_model_with_camera_positions(opt.cnet, opt.camera_models,
                                                  opt.image_files, estimated_camera_gcc);
    cameras_changed = true;
  }

  // If we have GPC's for pinhole cameras, try to do a simple affine
  // initialization of the camera parameters.
  // - This function also updates all the ControlNetwork world point
  //   positions.
  // - We could do this for other camera types too, but it would
  //   require us to be able to adjust our camera model positions.
  //   Otherwise we could init the adjustment values.
  if (opt.gcp_files.size() > 0) {
    if ((opt.camera_type == BaCameraType_Pinhole) && 
        !have_est_camera_positions) {
      if (opt.transform_cameras_using_gcp) {
        asp::transform_cameras_with_indiv_image_gcp(opt.cnet, opt.camera_models);
        cameras_changed = true;
      } else if (opt.transform_cameras_with_shared_gcp) {
        asp::transform_cameras_with_shared_gcp(opt.cnet, opt.camera_models);
            cameras_changed = true;
      } else if (opt.init_camera_using_gcp) {
        asp::init_camera_using_gcp(opt.cnet, opt.camera_models);
            cameras_changed = true;
      }
    }
    
    // Issue a warning if the GCPs are far away from the camera coordinates.
    // Do it only if the cameras did not change, as otherwise the cnet is outdated.
    if (!cameras_changed) 
      check_gcp_dists(opt.camera_models, opt.cnet, opt.forced_triangulation_distance);
  }
  
  int num_points = cnet.size();
  const int num_cameras = opt.image_files.size();

  // This is important to prevent a crash later
  if (num_points == 0 && !opt.apply_initial_transform_only) {
    vw_out() << "No points to optimize (GCP or otherwise). Cannot continue.\n";
    return;
  }
  
  // Create the storage arrays for the variables we will adjust.
  int num_lens_distortion_params = 0;
  if (opt.camera_type == BaCameraType_Pinhole) {
    boost::shared_ptr<vw::camera::PinholeModel> pinhole_ptr = 
            boost::dynamic_pointer_cast<vw::camera::PinholeModel>(opt.camera_models[0]);
    num_lens_distortion_params = pinhole_ptr->lens_distortion()->distortion_parameters().size();
    if (num_lens_distortion_params < 1) {
      // For the case where the camera has zero distortion parameters, use one dummy parameter
      //  just so we don't have to change the parameter block logic later on.
      num_lens_distortion_params = 1;
      opt.intrinisc_options.distortion_constant = true;
      opt.intrinisc_options.distortion_shared   = true;
    }
  }
  if (opt.camera_type == BaCameraType_OpticalBar) {
    num_lens_distortion_params = NUM_OPTICAL_BAR_EXTRA_PARAMS; // TODO: Share this constant!
  }
  asp::BAParams param_storage(num_points, num_cameras,
                               // Optical bar and pinhole are similar
                               opt.camera_type != BaCameraType_Other, 
                               // Must be the same for each pinhole camera
                               num_lens_distortion_params, 
                               opt.intrinisc_options);

  // Fill in the camera and intrinsic parameters.
  std::vector<boost::shared_ptr<camera::CameraModel>> new_cam_models;
  bool ans = false;
  switch (opt.camera_type) {
    case BaCameraType_Pinhole:
      ans = init_cams_pinhole(opt, param_storage, new_cam_models); break;
    case BaCameraType_OpticalBar:
      ans = init_cams_optical_bar(opt, param_storage, new_cam_models); break;
    default:
      ans = init_cams(opt, param_storage, new_cam_models);
  };

  if (ans)
    cameras_changed = true;
  
  // Certain input options change the cameras inside init_cams and we
  // need to update the point coordinates for the new cameras. It is
  // ok to leave the original vector of camera models unchanged.
  
  // TODO(oalexan1): Building the control network twice looks like a
  // hack. Try to understand why the cameras can't be updated first,
  // then building the control network just once.
  if (!opt.apply_initial_transform_only && cameras_changed) {
    vw_out() <<"Updating the control network." << std::endl;
    cnet = ControlNetwork("Updated network"); // Wipe it all first
    /*bool success = */
    // Building the control network below may fail if there are only GCP,
    // but we will continue nevertheless.
    bool triangulate_control_points = true;
    vw::ba::build_control_network(triangulate_control_points,
                                  cnet, new_cam_models,
                                  opt.image_files,
                                  opt.match_files,
                                  opt.min_matches,
                                  opt.min_triangulation_angle*(M_PI/180.0),
                                  opt.forced_triangulation_distance,
                                  opt.max_pairwise_matches);
    
    // Restore the rest of the cnet object
    num_gcp = vw::ba::add_ground_control_points(cnet, opt.gcp_files, opt.datum);
    
    check_gcp_dists(new_cam_models, opt.cnet, opt.forced_triangulation_distance);
    
    // Must update the number of points after the control network is recomputed
    num_points = cnet.size();
    param_storage.get_point_vector().resize(num_points*asp::BAParams::PARAMS_PER_POINT);
  }

  // Fill in the point vector with the starting values.
  for (int ipt = 0; ipt < num_points; ipt++)
    param_storage.set_point(ipt, cnet[ipt].position());

  // The camera positions and orientations before we float them
  // - This includes modifications from any initial transforms that were specified.
  asp::BAParams orig_parameters(param_storage);

  bool has_datum = (opt.datum.name() != asp::UNSPECIFIED_DATUM);
  if (has_datum && (opt.stereo_session == "pinhole") || 
      (opt.stereo_session == "nadirpinhole")) 
    saveCameraReport(opt, param_storage,  opt.datum, "initial");
    
  // TODO(oalexan1): Is it possible to avoid using CRNs?
  CRNJ crn;
  crn.from_cnet(cnet);

  if (opt.num_ba_passes <= 0)
    vw_throw(ArgumentErr() << "Error: Expecting at least one bundle adjust pass.\n");
  
  double final_cost;
  for (int pass = 0; pass < opt.num_ba_passes; pass++) {

    if (opt.apply_initial_transform_only)
      continue;
      
    vw_out() << "--> Bundle adjust pass: " << pass << std::endl;

    bool first_pass = (pass == 0);
    bool convergence_reached = true; // will change
    do_ba_ceres_one_pass(opt, crn, first_pass,
                         param_storage, orig_parameters,
                         convergence_reached, final_cost);
    
    int num_points_remaining = num_points - param_storage.get_num_outliers();
    if (num_points_remaining < opt.min_matches && num_gcp == 0) {
      // Do not throw if there exist gcp, as maybe that's all there is, and there
      // can be just a few of them.
      vw_throw(ArgumentErr() << "Error: Too few points remain after filtering!.\n");
    }
  } // End loop through passes

  double best_cost = final_cost;
  boost::shared_ptr<asp::BAParams> best_params_ptr(new asp::BAParams(param_storage));

  // This flow is only kicked in if opt.num_random_passes is positive, which
  // is not the default.
  std::string orig_out_prefix = opt.out_prefix;
  for (int pass = 0; pass < opt.num_random_passes; pass++) {

    if (opt.apply_initial_transform_only)
      continue;
    
    vw_out() << "\n--> Running bundle adjust pass " << pass 
             << " with random initial parameter offsets.\n";

    // Randomly distort the original inputs.
    param_storage.randomize_cameras();
    if (opt.solve_intrinsics)
      param_storage.randomize_intrinsics(opt.intrinsics_limits); // This handles sharing, etc.

    // Write output files to a temporary prefix
    opt.out_prefix = orig_out_prefix + "_rand";

    // Do another pass of bundle adjustment.
    bool first_pass = true; // this needs more thinking
    bool convergence_reached = true;
    do_ba_ceres_one_pass(opt, crn, first_pass,
                         param_storage, orig_parameters,
                         convergence_reached, final_cost);
    
    // Record the parameters of the best result.
    if (final_cost < best_cost) {
      vw_out() << "  --> Found a better solution!\n\n";
      best_cost = final_cost;
      best_params_ptr.reset(new asp::BAParams(param_storage));

      // Get a list of all the files that were generated in the random step.
      std::vector<std::string> rand_files;
      get_files_with_prefix(opt.out_prefix, rand_files);

      // Replace the existing output files with them.
      for (size_t i=0; i<rand_files.size(); i++) {
        std::string new_path = rand_files[i];
        boost::replace_all(new_path, opt.out_prefix, orig_out_prefix);
        boost::filesystem::copy_file(rand_files[i], new_path,
                                     boost::filesystem::copy_option::overwrite_if_exists);
      }
    }

    // Clear out the extra files that were generated
    std::string cmd("rm -f " + opt.out_prefix + "*");
    vw_out() << "Deleting temporary files: " << cmd << std::endl;
    vw::exec_cmd(cmd.c_str());
  }
  opt.out_prefix = orig_out_prefix; // So the cameras are written to the expected paths.

  // Write the results to disk.
  saveResults(opt, *best_params_ptr);

  if (has_datum && (opt.stereo_session == "pinhole") || 
      (opt.stereo_session == "nadirpinhole")) 
    saveCameraReport(opt, *best_params_ptr, opt.datum, "final");
  
} // end do_ba_ceres

/// Looks in the input camera position file to generate a GCC position for
/// each input camera.
/// - If no match is found, the coordinate is (0,0,0)
int load_estimated_camera_positions(Options &opt,
                                    std::vector<Vector3> & estimated_camera_gcc) {
  estimated_camera_gcc.clear();
  if (opt.camera_position_file == "")
    return 0;
  
  // Read the input csv file
  asp::CsvConv conv;
  conv.parse_csv_format(opt.csv_format_str, opt.csv_proj4_str);
  std::list<asp::CsvConv::CsvRecord> pos_records;
  typedef std::list<asp::CsvConv::CsvRecord>::const_iterator RecordIter;
  conv.read_csv_file(opt.camera_position_file, pos_records);

  // Set up a GeoReference object using the datum
  vw::cartography::GeoReference geo;
  geo.set_datum(opt.datum); // We checked for a datum earlier
  // Use user's csv_proj4 string, if provided, to add info to the georef.
  conv.parse_georef(geo);

  // For each input camera, find the matching position in the record list
  const int num_cameras = opt.image_files.size();
  estimated_camera_gcc.resize(num_cameras);
  
  const RecordIter no_match = pos_records.end();
  int num_matches_found = 0;
  for (int i=0; i<num_cameras; i++) {

    // Search for this image file in the records
    std::string file_name = opt.image_files[i];
    RecordIter iter;
    for (iter=pos_records.begin(); iter!=pos_records.end(); iter++) {
      // Match if the string in the file is contained in the input image string.
      // - May need to play around with this in the future!
      std::string field = iter->file;
      if (file_name.find(field) != std::string::npos) {
        estimated_camera_gcc[i] = conv.csv_to_cartesian(*iter, geo);
        break; // Match found, stop the iterator here.
      }
    }
    if (iter == no_match) {
      vw_out() << "WARNING: Camera file " << file_name << " not found in camera position file.\n";
      estimated_camera_gcc[i] = Vector3(0,0,0);
    }else
      ++num_matches_found;
  } // End loop to find position record for each camera

  return num_matches_found;  
}

void handle_arguments(int argc, char *argv[], Options& opt) {
  const double nan = std::numeric_limits<double>::quiet_NaN();
  std::string intrinsics_to_float_str, intrinsics_to_share_str,
    intrinsics_limit_str;
  bool  inline_adjustments;
  int   max_iterations_tmp;
  po::options_description general_options("");
  general_options.add_options()
    ("output-prefix,o",  po::value(&opt.out_prefix), "Prefix for output filenames.")
    ("cost-function",    po::value(&opt.cost_function)->default_value("Cauchy"),
     "Choose a cost function from: Cauchy, PseudoHuber, Huber, L1, L2, Trivial.")
    ("robust-threshold", po::value(&opt.robust_threshold)->default_value(0.5),
     "Set the threshold for robust cost functions. Increasing this makes the solver focus harder on the larger errors.")
    ("inline-adjustments",   po::bool_switch(&inline_adjustments)->default_value(false),
     "If this is set, and the input cameras are of the pinhole or panoramic type, apply the adjustments directly to the cameras, rather than saving them separately as .adjust files.")
    ("approximate-pinhole-intrinsics", po::bool_switch(&opt.approximate_pinhole_intrinsics)->default_value(false),
     "If it reduces computation time, approximate the lens distortion model.")
    ("solve-intrinsics",    po::bool_switch(&opt.solve_intrinsics)->default_value(false)->implicit_value(true),
     "Optimize intrinsic camera parameters.  Only used for pinhole cameras.")
    ("intrinsics-to-float", po::value(&intrinsics_to_float_str)->default_value(""),
     "If solving for intrinsics and desired to float only a few of them, specify here, in quotes, one or more of: focal_length, optical_center, other_intrinsics. Not specifying anything will float all of them.")
    ("intrinsics-to-share", po::value(&intrinsics_to_share_str)->default_value(""),
     "If solving for intrinsics and desired to share only a few of them, specify here, in quotes, one or more of: focal_length, optical_center, other_intrinsics. Not specifying anything, will share none of them.")
    ("intrinsics-limits", 
     po::value(&intrinsics_limit_str)->default_value(""),
     "Specify minimum and maximum ratios for the intrinsic parameters. Values must be in min max pairs and are applied in the order [focal length, optical center, other intrinsics] until all of the limits are used. Check the documentation to dermine how many intrinsic parameters are used for your cameras.")
    ("camera-positions",    po::value(&opt.camera_position_file)->default_value(""),
     "Specify a csv file path containing the estimated positions of the input cameras.  Only used with the inline-adjustments option.")
    ("init-camera-using-gcp",  po::bool_switch(&opt.init_camera_using_gcp)->default_value(false)->implicit_value(true),
     "Given an image, a pinhole camera lacking correct position and orientation, and a GCP file, find the pinhole camera with given intrinsics most consistent with the GCP.")
    ("transform-cameras-with-shared-gcp",  po::bool_switch(&opt.transform_cameras_with_shared_gcp)->default_value(false)->implicit_value(true),
    "Given at least 3 GCP, with each seen in at least 2 images, "
    "find the triangulated positions based on pixels values in the GCP, "
    "and apply a rotation + translation + scale transform to the entire "
    "camera system so that the triangulated points get mapped to the ground "
     "coordinates in the GCP.")
    ("transform-cameras-using-gcp",  po::bool_switch(&opt.transform_cameras_using_gcp)->default_value(false)->implicit_value(true),
     "Given a set of GCP, with at least two images having at least three GCP each (but with each GCP not shared among the images), transform the cameras to ground coordinates. This is not as robust as --transform-cameras-with-shared-gcp.")
    ("disable-pinhole-gcp-init",  po::bool_switch(&opt.disable_pinhole_gcp_init)->default_value(false)->implicit_value(true),
     "Do not try to initialize the positions of pinhole cameras based on input GCPs. This ignored as is now the default. See also: --init-camera-using-gcp.")
    ("input-adjustments-prefix",  po::value(&opt.input_prefix),
     "Prefix to read initial adjustments from, written by a previous invocation of this program.")
    ("initial-transform",   po::value(&opt.initial_transform_file)->default_value(""),
     "Before optimizing the cameras, apply to them the 4x4 rotation + translation transform from this file. The transform is in respect to the planet center, such as written by pc_align's source-to-reference or reference-to-source alignment transform. Set the number of iterations to 0 to stop at this step. If --input-adjustments-prefix is specified, the transform gets applied after the adjustments are read.")
    ("fixed-camera-indices",    po::value(&opt.fixed_cameras_indices_str)->default_value(""),
     "A list of indices, in quotes and starting from 0, with space as separator, corresponding to cameras to keep fixed during the optimization process.")
    ("fixed-image-list",    po::value(&opt.fixed_image_list)->default_value(""),
     "A file having a list of images (separated by spaces or newlines) whose cameras should be fixed during optimization.")
    ("fix-gcp-xyz",       po::bool_switch(&opt.fix_gcp_xyz)->default_value(false)->implicit_value(true),
     "If the GCP are highly accurate, use this option to not float them during the optimization.")

    ("csv-format",        po::value(&opt.csv_format_str)->default_value(""), asp::csv_opt_caption().c_str())
    ("csv-proj4",         po::value(&opt.csv_proj4_str)->default_value(""),
     "The PROJ.4 string to use to interpret the entries in input CSV files.")
    ("reference-terrain", po::value(&opt.reference_terrain)->default_value(""),
     "An externally provided trustworthy 3D terrain, either as a DEM or as a lidar file, very close (after alignment) to the stereo result from the given images and cameras that can be used as a reference, instead of GCP, to optimize the intrinsics of the cameras.")
    ("max-num-reference-points", po::value(&opt.max_num_reference_points)->default_value(100000000),
     "Maximum number of (randomly picked) points from the reference terrain to use.")
    ("disparity-list",           po::value(&opt.disparity_list)->default_value(""),
     "The unaligned disparity files to use when optimizing the intrinsics based on a reference terrain. Specify them as a list in quotes separated by spaces. First file is for the first two images, second is for the second and third images, etc. If an image pair has no disparity file, use 'none'.")
    ("max-disp-error",           po::value(&opt.max_disp_error)->default_value(-1),
     "When using a reference terrain as an external control, ignore as outliers xyz points which projected in the left image and transported by disparity to the right image differ by the projection of xyz in the right image by more than this value in pixels.")
    ("reference-terrain-weight", po::value(&opt.reference_terrain_weight)->default_value(1.0),
     "How much weight to give to the cost function terms involving the reference terrain.")
    ("heights-from-dem",   po::value(&opt.heights_from_dem)->default_value(""),
     "If the cameras have already been bundle-adjusted and aligned to a known high-quality DEM, "
     "in the triangulated xyz points replace the heights with the ones from this DEM, and "
     "fix those points unless --heights-from-dem-weight is positive.")
    ("heights-from-dem-weight", po::value(&opt.heights_from_dem_weight)->default_value(1.0),
     "How much weight to give to keep the triangulated points close to the DEM if specified via "
     "--heights-from-dem. If the weight is not positive, keep the triangulated points fixed. "
     "This value should be inversely proprortional with ground sample distance, as "
     "then it will convert the measurements from meters to pixels, which is consistent "
     "with the reprojection error term.")
    ("heights-from-dem-robust-threshold",
     po::value(&opt.heights_from_dem_robust_threshold)->default_value(0.5),
     "If positive, this is the robust threshold to use keep the triangulated points "
     "close to the DEM if specified via --heights-from-dem. This is applied after the "
     "point differences are multiplied by --heights-from-dem-weight. It should help with "
     "attenuating large height difference outliers.")
    ("mapproj-dem", po::value(&opt.mapproj_dem)->default_value(""),
     "If specified, mapproject every pair of matched interest points onto this DEM and compute "
     "their distance, then percentiles of such distances for each image pair and "
     "for each image vs the rest. "
     "This is done after bundle adjustment and outlier removal. "
     "Measured in meters.")
    ("reference-dem",  po::value(&opt.ref_dem)->default_value(""),
     "If specified, intersect rays from matching pixels with this DEM, find the average, and constrain during optimization that rays keep on intersecting close to this point. This works even when the rays are almost parallel, but then consider using the option --forced-triangulation-distance. See also --reference-dem-weight and --reference-dem-robust-threshold.")
    ("reference-dem-weight", po::value(&opt.ref_dem_weight)->default_value(1.0),
     "Multiply the xyz differences for the --reference-dem option by this weight.")
    ("reference-dem-robust-threshold", po::value(&opt.ref_dem_robust_threshold)->default_value(0.5),
     "Use this robust threshold for the weighted xyz differences with the --reference-dem option.")
    ("datum",            po::value(&opt.datum_str)->default_value(""),
     "Use this datum. Needed only for ground control points, a camera position file, or for RPC sessions. Options: WGS_1984, D_MOON (1,737,400 meters), D_MARS (3,396,190 meters), MOLA (3,396,000 meters), NAD83, WGS72, and NAD27. Also accepted: Earth (=WGS_1984), Mars (=D_MARS), Moon (=D_MOON).")
    ("semi-major-axis",  po::value(&opt.semi_major)->default_value(0),
     "Explicitly set the datum semi-major axis in meters (see above).")
    ("semi-minor-axis",  po::value(&opt.semi_minor)->default_value(0),
     "Explicitly set the datum semi-minor axis in meters (see above).")
    ("session-type,t",   po::value(&opt.stereo_session)->default_value(""),
     "Select the stereo session type to use for processing. Usually the program can select this automatically by the file extension, except for xml cameras. See the doc for options.")
    ("min-matches",      po::value(&opt.min_matches)->default_value(30),
     "Set the minimum  number of matches between images that will be considered.")
    ("max-pairwise-matches", po::value(&opt.max_pairwise_matches)->default_value(10000),
     "Reduce the number of matches per pair of images to at most this "
     "number, by selecting a random subset, if needed. This happens "
     "when setting up the optimization, and before outlier filtering.")
    ("ip-detect-method", po::value(&opt.ip_detect_method)->default_value(0),
     "Interest point detection algorithm (0: Integral OBALoG (default), 1: OpenCV SIFT, 2: OpenCV ORB.")
    ("epipolar-threshold",      po::value(&opt.epipolar_threshold)->default_value(-1),
     "Maximum distance from the epipolar line to search for IP matches. Default: automatic calculation. A higher values will result in more matches.")
    ("ip-inlier-factor",        po::value(&opt.ip_inlier_factor)->default_value(0.2),
     "A higher factor will result in more interest points, but perhaps also more outliers. This is used only with homography alignment, such as for the pinhole session.")
    ("ip-uniqueness-threshold", po::value(&opt.ip_uniqueness_thresh)->default_value(0.8),
     "A higher threshold will result in more interest points, but perhaps less unique ones.")
    ("ip-side-filter-percent",  po::value(&opt.ip_edge_buffer_percent)->default_value(-1),
     "Remove matched IPs this percentage from the image left/right sides.")
    ("normalize-ip-tiles", 
     po::bool_switch(&opt.ip_normalize_tiles)->default_value(false)->implicit_value(true),
     "Individually normalize tiles used for IP detection.")
    ("num-obalog-scales",      po::value(&opt.num_scales)->default_value(-1),
     "How many scales to use if detecting interest points with OBALoG. If not specified, 8 will be used. More can help for images with high frequency artifacts.")
    ("nodata-value",           po::value(&opt.nodata_value)->default_value(nan),
     "Pixels with values less than or equal to this number are treated as no-data. This overrides the no-data values from input images.")
    ("num-iterations",       po::value(&opt.num_iterations)->default_value(1000),
     "Set the maximum number of iterations.") 
    ("max-iterations",       po::value(&max_iterations_tmp)->default_value(1000),
     "Set the maximum number of iterations.") // alias for num-iterations
    ("parameter-tolerance",  po::value(&opt.parameter_tolerance)->default_value(1e-8),
     "Stop when the relative error in the variables being optimized is less than this.")
    ("overlap-limit",        po::value(&opt.overlap_limit)->default_value(0),
     "Limit the number of subsequent images to search for matches to the current image to this value. By default match all images.")
    ("overlap-list",         po::value(&opt.overlap_list_file)->default_value(""),
     "A file containing a list of image pairs, one pair per line, separated by a space, which are expected to overlap. Matches are then computed only among the images in each pair.")
    ("auto-overlap-params",  po::value(&opt.auto_overlap_params)->default_value(""),
     "Determine which camera images overlap by finding the lon-lat bounding boxes "
     "of their footprints given the specified DEM, expanding them by a given percentage, "
     "and see if those intersect. A higher percentage should be used when there is more "
     "uncertainty about the input camera poses. Example: 'dem.tif 15'.")
    ("auto-overlap-buffer",  po::value(&opt.auto_overlap_buffer)->default_value(-1.0),
     "Try to automatically determine which images overlap. Used only if "
     "this option is explicitly set. Only supports Worldview style XML "
     "camera files. The lon-lat footprints of the cameras are expanded "
     "outwards on all sides by this value (in degrees), before checking "
     "if they intersect.")
    ("image-list", po::value(&opt.image_list)->default_value(""),
     "A file containing the list of images, when they are too many to specify on the command line. Use space or newline as separator. See also --camera-list and --mapprojected-data-list.")
    ("camera-list", po::value(&opt.camera_list)->default_value(""),
     "A file containing the list of cameras, when they are too many to specify on the command "
     "line. If the images have embedded camera information, such as for ISIS, this file must "
     "be empty but must be specified if --image-list is specified.")
    ("mapprojected-data-list", po::value(&opt.mapprojected_data_list)->default_value(""),
     "A file containing the list of mapprojected images and the DEM (see --mapprojected-data), when they are too many to specify on the command line.")
    ("position-filter-dist", po::value(&opt.position_filter_dist)->default_value(-1),
     "Set a distance in meters and don't perform IP matching on images with an estimated camera center farther apart than this distance.  Requires --camera-positions.")
    ("match-first-to-last", po::value(&opt.match_first_to_last)->default_value(false)->implicit_value(true),
     "Match first several images to last several images by extending the logic of --overlap-limit past the last image to the earliest ones.")
    
    ("rotation-weight",      po::value(&opt.rotation_weight)->default_value(0.0),
     "A higher weight will penalize more rotation deviations from the original configuration.")
    ("translation-weight",   po::value(&opt.translation_weight)->default_value(0.0),
     "A higher weight will penalize more translation deviations from the original configuration.")
    ("camera-weight",        po::value(&opt.camera_weight)->default_value(1.0),
     "The weight to give to the constraint that the camera positions/orientations stay close to the original values. A higher weight means that the values will change less. The options --rotation-weight and --translation-weight can be used for finer-grained control.")
    ("tri-weight", po::value(&opt.tri_weight)->default_value(0.0),
     "The weight to give to the constraint that optimized triangulated "
     "points stay close to original triangulated points. A positive value will help "
     "ensure the cameras do not move too far, but a large value may prevent convergence. "
     "It is suggested to use here 0.1 to 0.5 divided by image gsd. Use it together with "
     "--tri-robust-threshold. Does not apply to GCP or points constrained by a DEM. "
     "Set --camera-weight to 0 when using this.")
    ("tri-robust-threshold",
     po::value(&opt.tri_robust_threshold)->default_value(0.1),
     "Use this robust threshold to attenuate large differences "
     "between initial and optimized triangulation points, after multiplying them by --tri-weight.")
    ("overlap-exponent",     po::value(&opt.overlap_exponent)->default_value(0.0),
     "If a feature is seen in n >= 2 images, give it a weight proportional with (n-1)^exponent.")
    ("ip-per-tile",          po::value(&opt.ip_per_tile)->default_value(0),
      "How many interest points to detect in each 1024^2 image tile (default: automatic determination). This is before matching. Not all interest points will have a match. See also --matches-per-tile.")
    ("ip-per-image",              po::value(&opt.ip_per_image)->default_value(0),
     "How many interest points to detect in each image (default: automatic determination). It is overridden by --ip-per-tile if provided.")
    ("num-passes",           po::value(&opt.num_ba_passes)->default_value(2),
     "How many passes of bundle adjustment to do, with given number of iterations in each pass. For more than one pass, outliers will be removed between passes using --remove-outliers-params, and re-optimization will take place. Residual files and a copy of the match files with the outliers removed (*-clean.match) will be written to disk.")
    ("num-random-passes",           po::value(&opt.num_random_passes)->default_value(0),
     "After performing the normal bundle adjustment passes, do this many more passes using the same matches but adding random offsets to the initial parameter values with the goal of avoiding local minima that the optimizer may be getting stuck in.")
    ("remove-outliers-params", 
     po::value(&opt.remove_outliers_params_str)->default_value("75.0 3.0 2.0 3.0", "'pct factor err1 err2'"),
     "Outlier removal based on percentage, when more than one bundle adjustment pass is used. Triangulated points (that are not GCP) with reprojection error in pixels larger than min(max('pct'-th percentile * 'factor', err1), err2) will be removed as outliers. Hence, never remove errors smaller than err1 but always remove those bigger than err2. Specify as a list in quotes. Also remove outliers based on distribution of interest point matches and triangulated points. Default: '75.0 3.0 2.0 3.0'.")
    ("elevation-limit",        po::value(&opt.elevation_limit)->default_value(Vector2(0,0), "auto"),
     "Remove as outliers interest points (that are not GCP) for which the elevation of the triangulated position (after cameras are optimized) is outside of this range. Specify as two values: min max.")
    // Note that we count later on the default for lon_lat_limit being BBox2(0,0,0,0).
    ("lon-lat-limit",          po::value(&opt.lon_lat_limit)->default_value(BBox2(0,0,0,0), "auto"),
     "Remove as outliers interest points (that are not GCP) for which the longitude and latitude of the triangulated position (after cameras are optimized) are outside of this range. Specify as: min_lon min_lat max_lon max_lat.")
    ("match-files-prefix",  po::value(&opt.match_files_prefix)->default_value(""),
     "Use the match files from this prefix instead of the current output prefix. This implies --skip-matching.")
    ("clean-match-files-prefix",  po::value(&opt.clean_match_files_prefix)->default_value(""),
     "Use as input match files the *-clean.match files from this prefix. This implies --skip-matching.")
    ("enable-rough-homography",
     po::bool_switch(&opt.enable_rough_homography)->default_value(false)->implicit_value(true),
     "Enable the step of performing datum-based rough homography for interest point matching. This is best used with reasonably reliable input cameras and a wide footprint on the ground.")
    ("skip-rough-homography",
     po::bool_switch(&opt.skip_rough_homography)->default_value(false)->implicit_value(true),
     "Skip the step of performing datum-based rough homography. This obsolete option is ignored as is the default.")
    ("enable-tri-ip-filter",
     po::bool_switch(&opt.enable_tri_filtering)->default_value(false)->implicit_value(true),
     "Enable triangulation-based interest points filtering. This is best used with reasonably reliable input cameras.")
    ("disable-tri-ip-filter",
     po::bool_switch(&opt.disable_tri_filtering)->default_value(false)->implicit_value(true),
     "Disable triangulation-based interest points filtering. This obsolete option is ignored as is the default.")
    ("no-datum", po::bool_switch(&opt.no_datum)->default_value(false)->implicit_value(true),
     "Do not assume a reliable datum exists, such as for irregularly shaped bodies.")
    ("individually-normalize", 
     po::bool_switch(&opt.individually_normalize)->default_value(false)->implicit_value(true),
     "Individually normalize the input images instead of using common values.")
    ("ip-triangulation-max-error",  po::value(&opt.ip_triangulation_max_error)->default_value(-1),
     "When matching IP, filter out any pairs with a triangulation error higher than this.")
    ("ip-num-ransac-iterations", po::value(&opt.ip_num_ransac_iterations)->default_value(1000),
     "How many RANSAC iterations to do in interest point matching.")
    ("min-triangulation-angle", po::value(&opt.min_triangulation_angle)->default_value(0.1),
     "A triangulated point will be accepted as valid only if at "
     "least two of the rays which converge at it have a triangulation "
     "angle of at least this (measured in degrees).")
    ("forced-triangulation-distance",      po::value(&opt.forced_triangulation_distance)->default_value(-1),
     "When triangulation fails, for example, when input cameras are inaccurate, artificially create a triangulation point this far ahead of the camera, in units of meter.")
    ("use-lon-lat-height-gcp-error",
     po::bool_switch(&opt.use_llh_error)->default_value(false)->implicit_value(true),
     "When having GCP, interpret the three standard deviations in the GCP file as applying not to x, y, and z, but rather to latitude, longitude, and height.")
    ("enable-correct-velocity-aberration", po::bool_switch(&opt.enable_correct_velocity_aberration)->default_value(false)->implicit_value(true),
     "Turn on velocity aberration correction for Optical Bar and non-ISIS linescan cameras. This option impairs the convergence of bundle adjustment.")
    ("enable-correct-atmospheric-refraction", po::bool_switch(&opt.enable_correct_atmospheric_refraction)->default_value(false)->implicit_value(true),
     "Turn on atmospheric refraction correction for Optical Bar and non-ISIS linescan cameras. This option impairs the convergence of bundle adjustment.")
    ("dg-use-csm", po::bool_switch(&opt.dg_use_csm)->default_value(false)->implicit_value(true),
     "Use the CSM model with DigitalGlobe linescan cameras (-t dg). No corrections are done for velocity aberration or atmospheric refraction.")
    ("mapprojected-data",  po::value(&opt.mapprojected_data)->default_value(""),
     "Given map-projected versions of the input images and the DEM they "
     "were mapprojected onto, create interest point matches among the  "
     "mapprojected images, unproject and save those matches, then  "
     "continue with bundle adjustment. Existing match files will be  "
     "reused. Specify the mapprojected images and the DEM as a string in  "
     "quotes, separated by spaces. An example is in the documentation.")
    ("matches-per-tile",  po::value(&opt.matches_per_tile)->default_value(0),
     "How many interest point matches to compute in each image tile (of size "
      "normally 1024^2 pixels). Use a value of --ip-per-tile a few times larger "
      "than this. See also --matches-per-tile-params.")
    ("save-cnet-as-csv", po::bool_switch(&opt.save_cnet_as_csv)->default_value(false)->implicit_value(true),
     "Save the control network containing all interest points in the format used by ground control points, so it can be inspected.")
    ("gcp-from-mapprojected-images", po::value(&opt.gcp_from_mapprojected)->default_value(""),
     "Given map-projected versions of the input images, the DEM the were mapprojected onto, and interest point matches among all of these created in stereo_gui, create GCP for the input images to align them better to the DEM. This is experimental and not documented.")
    ("instance-count",      po::value(&opt.instance_count)->default_value(1),
     "The number of bundle_adjustment processes being run in parallel.")
    ("instance-index",      po::value(&opt.instance_index)->default_value(0),
     "The index of this parallel bundle adjustment process.")
    ("stop-after-statistics",    po::bool_switch(&opt.stop_after_stats)->default_value(false)->implicit_value(true),
     "Quit after computing image statistics.")
    ("stop-after-matching",    po::bool_switch(&opt.stop_after_matching)->default_value(false)->implicit_value(true),
     "Quit after writing all match files.")
    ("force-reuse-match-files", po::bool_switch(&opt.force_reuse_match_files)->default_value(false)->implicit_value(true),
     "Force reusing the match files even if older than the images or cameras.")
    ("skip-matching",    po::bool_switch(&opt.skip_matching)->default_value(false)->implicit_value(true),
     "Only use image matches which can be loaded from disk. This implies --force-reuse-match-files.")
    ("save-intermediate-cameras", po::value(&opt.save_intermediate_cameras)->default_value(false)->implicit_value(true),
     "Save the values for the cameras at each iteration.")
    ("apply-initial-transform-only", po::value(&opt.apply_initial_transform_only)->default_value(false)->implicit_value(true),
     "Apply to the cameras the transform given by --initial-transform. "
     "No iterations, GCP loading, image matching, or report generation "
     "take place. Using --num-iterations 0 and without this option "
     "will create those.")
    ("proj-win", po::value(&opt.proj_win)->default_value(BBox2(0,0,0,0), "auto"),
     "Flag as outliers input triangulated points not in this proj win (box in projected units as provided by --proj_str). This should be generous if the input cameras have significant errors.")
    ("proj-str",   po::value(&opt.proj_str)->default_value(""),
     "To be used in conjunction with --proj_win.")
    ("matches-per-tile-params",  po::value(&opt.matches_per_tile_params)->default_value(Vector2(1024, 1280), "1024 1280"),
     "To be used with --matches-per-tile. The first value is the image tile "
      "size for both images. A larger second value allows each right tile to "
      "further expand to this size, resulting in the tiles overlapping. This may be "
      "needed if the homography alignment between these images is not great, as "
      "this transform is used to pair up left and right image tiles.")
    ("save-vwip",    po::bool_switch(&opt.save_vwip)->default_value(false)->implicit_value(true),
     "Save .vwip files (intermediate files for creating .match files). For parallel_bundle_adjust these will be saved in subdirectories, as they depend on the image pair. Must start with an empty output directory for this to work.")
    ("vwip-prefix",  po::value(&opt.vwip_prefix),
     "Save .vwip files with this prefix. This is a private option used by parallel_bundle_adjust.")
    ("ip-debug-images",        po::value(&opt.ip_debug_images)->default_value(false)->implicit_value(true),
     "Write debug images to disk when detecting and matching interest points.");
    
  general_options.add(vw::GdalWriteOptionsDescription(opt));

  po::options_description positional("");
  positional.add_options()
    ("input-files", po::value(&opt.image_files));

  po::positional_options_description positional_desc;
  positional_desc.add("input-files", -1);

  std::string usage("<images> <cameras> <optional ground control points> -o <output prefix> [options]");
  bool allow_unregistered = false;
  std::vector<std::string> unregistered;
  po::variables_map vm =
    asp::check_command_line(argc, argv, opt, general_options, general_options,
                            positional, positional_desc, usage,
                             allow_unregistered, unregistered);

  // Separate out GCP files
  opt.gcp_files = asp::get_files_with_ext(opt.image_files, ".gcp", true);
  const size_t num_gcp_files = opt.gcp_files.size();
  vw_out() << "Found " << num_gcp_files << " GCP files on the command line.\n";

  // Separate the cameras from the images
  std::vector<std::string> inputs = opt.image_files;

  if (!opt.image_list.empty()) {
    // Read the images and cameras and put them in 'inputs' to be parsed later
    if (opt.camera_list.empty())
      vw_throw(ArgumentErr()
               << "The option --image-list must be invoked together with --camera-list.\n");
    if (!inputs.empty())
      vw_throw(ArgumentErr() << "The option --image-list was specified, but also "
               << "images or cameras on the command line.\n");
    asp::read_list(opt.image_list, inputs);
    std::vector<std::string> tmp;
    asp::read_list(opt.camera_list, tmp);
    for (size_t it = 0; it < tmp.size(); it++) 
      inputs.push_back(tmp[it]);
  }

  // Sanity checks
  if ((!opt.camera_list.empty() || !opt.mapprojected_data_list.empty()) && opt.image_list.empty())
    vw_throw(ArgumentErr() << "Found --camera-list and --mapprojected-data-list, "
             << "but not --image-list.\n");
  if (!opt.mapprojected_data.empty() && !opt.mapprojected_data_list.empty())
    vw_throw(ArgumentErr() << "Cannot specify both --mapprojected-data and "
             << "--mapprojected-data-list.\n");
  
  bool ensure_equal_sizes = true;
  asp::separate_images_from_cameras(inputs,
                                    opt.image_files, opt.camera_files, // outputs
                                    ensure_equal_sizes); 
  
  asp::check_for_duplicates(opt.image_files, opt.camera_files, opt.out_prefix);
  
  // Sanity check
  if (opt.image_files.size() != (int)opt.camera_files.size()){
    vw_out() << "Detected " << opt.image_files.size() << " images and "
             << opt.camera_files.size() << " cameras.\n";
    vw_throw(ArgumentErr() << "Must have as many cameras as we have images.\n");
  }
  
  // TODO: Check for duplicates in opt.image_files!

  if (opt.image_files.empty())
    vw_throw( ArgumentErr() << "Missing input image files.\n"
                            << usage << general_options );

  // TODO(oalexan1): This duplicates logic from StereoSessionFactory.cc.
  // But need to ensure nothing breaks below.

  // Reusing match files implies that we skip matching
  if (opt.clean_match_files_prefix != "" || opt.match_files_prefix != "")
    opt.skip_matching = true;
  
  //  When skipping matching, we are already forced to reuse match
  //  files based on the logic in the code, but here enforce it
  //  explicitly anyway.
  if (opt.skip_matching) 
    opt.force_reuse_match_files = true;

  if (opt.auto_overlap_params != "" && opt.skip_matching) {
    vw_out() << "Ignoring --auto-overlap-params since no matching takes place.\n";
    opt.auto_overlap_params = "";
  }
  
  // Work out the camera model type to use
  boost::to_lower(opt.stereo_session);
  opt.camera_type = BaCameraType_Other;
  if (inline_adjustments) {

    // Try to guess the session 
    if (opt.stereo_session == ""){
      try {
        // If we can open a pinhole camera file, that means
        // we are good. We prefer nadirpinhole to pinhole
        // session.
        PinholeModel(opt.camera_files[0]);
        opt.stereo_session = "nadirpinhole";
      }catch(std::exception const& e){}
    }
    
    if ((opt.stereo_session == "pinhole") || 
        (opt.stereo_session == "nadirpinhole")) {
      opt.camera_type = BaCameraType_Pinhole;
    } else {
      if (opt.stereo_session == "opticalbar")
        opt.camera_type = BaCameraType_OpticalBar;
      else
        vw_throw( ArgumentErr() << "Cannot use inline adjustments with session: "
                  << opt.stereo_session << "\n"
                                << usage << general_options );
    }
  } // End resolving the model type
  
  if (opt.transform_cameras_using_gcp &&
      (!inline_adjustments) &&
      (opt.camera_type != BaCameraType_Pinhole)) {
    vw_throw( ArgumentErr() << "Transforming cameras using GCP works only for pinhole "
              << "cameras and with the --inline-adjustments flag.\n"
              << usage << general_options );
  }
  
  if (opt.overlap_list_file != "" && opt.overlap_limit > 0)
    vw_throw( ArgumentErr() << "Cannot specify both the overlap limit and the overlap list.\n"
              << usage << general_options );

  if (opt.overlap_list_file != "" && opt.match_first_to_last > 0)
    vw_throw( ArgumentErr() << "Cannot specify both the overlap limit and --match-first-to-last.\n"
              << usage << general_options );
    
  if (opt.overlap_limit < 0)
    vw_throw( ArgumentErr() << "Must allow search for matches between "
              << "at least each image and its subsequent one.\n" << usage << general_options );
  
  // By default, try to match all of the images!
  if (opt.overlap_limit == 0)
    opt.overlap_limit = opt.image_files.size();

  if (int(opt.overlap_list_file != "") + int(!vm["auto-overlap-buffer"].defaulted()) +
      int(opt.auto_overlap_params != "") > 1)
    vw_throw( ArgumentErr() << "Cannot specify more than one of --overlap-list, "
              << "--auto-overlap-params, and --auto-overlap-buffer.\n"
              << usage << general_options);

  opt.have_overlap_list = false;
  if (opt.overlap_list_file != "") {
   opt.have_overlap_list = true;
    if (!fs::exists(opt.overlap_list_file))
      vw_throw( ArgumentErr() << "The overlap list does not exist.\n" << usage
                << general_options );
    opt.overlap_list.clear();
    std::string image1, image2;
    std::ifstream ifs(opt.overlap_list_file.c_str());
    while (ifs >> image1 >> image2){
      opt.overlap_list.insert(std::make_pair(image1, image2));
      opt.overlap_list.insert(std::make_pair(image2, image1));
    }
    ifs.close();
  } else if (!vm["auto-overlap-buffer"].defaulted()) {
    opt.have_overlap_list = true;
    auto_build_overlap_list(opt, opt.auto_overlap_buffer);
  }
  // The third alternative, --auto-overlap-params will be handled when we have cameras
  
  if (opt.camera_weight < 0.0)
    vw_throw( ArgumentErr() << "The camera weight must be non-negative.\n" << usage
                            << general_options );

  if ( opt.rotation_weight < 0.0 )
    vw_throw( ArgumentErr() << "The rotation weight must be non-negative.\n" << usage
                            << general_options );

  if ( opt.translation_weight < 0.0 )
    vw_throw( ArgumentErr() << "The translation weight must be non-negative.\n" << usage
                            << general_options );

  if (opt.tri_weight < 0.0)
    vw_throw( ArgumentErr() << "The triangulation weight must be non-negative.\n" << usage
              << general_options );
  
  if (opt.tri_weight > 0 && opt.camera_weight > 0) 
    vw_throw( ArgumentErr() << "When --tri-weight is positive, set to zero "
              << "--camera-weight. Can use --rotation-weight and --translation-weight.\n");
  
  // NOTE(oalexan1): The reason min_triangulation_angle cannot be 0 is deep inside
  // StereoModel.cc. Better keep it this way than make too many changes there.
  if (opt.min_triangulation_angle <= 0.0)
    vw_throw( ArgumentErr() << "The minimum triangulation angle must be positive.\n");
  
  // TODO: Make sure the normal model loading catches this error.
  //if (opt.create_pinhole && !asp::has_pinhole_extension(opt.camera_files[0]))
  //  vw_throw( ArgumentErr() << "Cannot use special pinhole handling with non-pinhole input!\n");

  if ((opt.camera_type == BaCameraType_Other) && opt.solve_intrinsics)
    vw_throw( ArgumentErr() << "Solving for intrinsic parameters is only supported with "
              << "pinhole and optical bar cameras.\n");

  if ((opt.camera_type!=BaCameraType_Pinhole) && opt.approximate_pinhole_intrinsics)
    vw_throw( ArgumentErr() << "Cannot approximate intrinsics unless using pinhole cameras.\n");

  if (opt.approximate_pinhole_intrinsics && opt.solve_intrinsics)
    vw_throw( ArgumentErr() << "Cannot approximate intrinsics while solving for them.\n");

  if (opt.camera_type != BaCameraType_Other &&
      opt.camera_type != BaCameraType_Pinhole &&
      opt.input_prefix != "")
    vw_throw( ArgumentErr() << "Can only use initial adjustments with camera type "
              << "'other' or 'pinhole'. Here likely having optical bar cameras.\n");

  vw::string_replace(opt.remove_outliers_params_str, ",", " "); // replace any commas
  opt.remove_outliers_params = vw::str_to_vec<vw::Vector<double, 4>>(opt.remove_outliers_params_str);
  
  // Ensure good order
  if ( opt.lon_lat_limit != BBox2(0,0,0,0) ) {
    if ( opt.lon_lat_limit.min().y() > opt.lon_lat_limit.max().y() ) 
      std::swap( opt.lon_lat_limit.min().y(), opt.lon_lat_limit.max().y() );
    if ( opt.lon_lat_limit.min().x() > opt.lon_lat_limit.max().x() ) 
      std::swap( opt.lon_lat_limit.min().x(), opt.lon_lat_limit.max().x() );
  }
  
  if (!opt.camera_position_file.empty() && opt.csv_format_str == "")
    vw_throw( ArgumentErr() << "When using a camera position file, the csv-format "
              << "option must be set.\n");

  if (opt.max_pairwise_matches <= 0) 
    vw_throw( ArgumentErr() << "Must have a positive number of max pairwise matches.\n");
  
  // Copy the IP settings to the global stereo_settings() object
  opt.copy_to_asp_settings();

  // Try to infer the datum, if possible, from the images. For
  // example, Cartosat-1 has that info in the Tif file.
  bool guessed_datum = false;
  if (opt.datum_str == "") {
    vw::cartography::GeoReference georef;
    for (size_t it = 0; it < opt.image_files.size(); it++) {
      bool is_good = vw::cartography::read_georeference(georef, opt.image_files[it]);
      if (is_good){
        opt.datum = georef.datum();
        opt.datum_str = opt.datum.name();
        guessed_datum = true;
      }
    }
  }

  // Try to infer the datum from the reference terrain
  if (opt.reference_terrain != "") {
    std::string file_type = asp::get_cloud_type(opt.reference_terrain);
    if (file_type == "DEM") {
      vw::cartography::GeoReference georef;
      bool is_good = vw::cartography::read_georeference(georef, opt.reference_terrain);
      if (!is_good)
        vw_throw( ArgumentErr() << "The reference terrain DEM does not have a georeference.\n");
      if (opt.datum_str == ""){
        opt.datum = georef.datum();
        opt.datum_str = opt.datum.name();
        guessed_datum = true;
      }
    }
  }

  if (opt.robust_threshold <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --robust-threshold must be positive.\n");

  if (opt.tri_robust_threshold <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --tri-robust-threshold must be positive.\n");

  if ((!opt.heights_from_dem.empty() || !opt.ref_dem.empty()) && opt.fix_gcp_xyz)
    vw_throw(ArgumentErr()
             << "The option --fix-gcp-xyz is not compatible with a DEM constraint.\n");
  
  if (!opt.heights_from_dem.empty() && !opt.ref_dem.empty()) 
    vw_throw(ArgumentErr() << "Cannot specify more than one of: --heights-from-dem "
             << "and --reference-dem.\n");

  if (opt.heights_from_dem_weight < 0.0) 
    vw_throw(ArgumentErr() << "The value of --heights-from-dem-weight must be non-negative.\n");
  
  if (opt.heights_from_dem_robust_threshold < 0.0) 
    vw_throw(ArgumentErr() << "The value of --heights-from-robust-threshold must be non-negative.\n");

  if (opt.ref_dem_weight <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --reference-dem-weight must be positive.\n");
  
  if (opt.ref_dem_robust_threshold <= 0.0) 
    vw_throw(ArgumentErr() << "The value of --reference-dem-robust-threshold must be positive.\n");

  bool have_dem = (!opt.heights_from_dem.empty() || !opt.ref_dem.empty());
  
  // Try to infer the datum from the heights-from-dem
  std::string dem_file;
  if (opt.heights_from_dem != "") 
    dem_file = opt.heights_from_dem;
  else if (opt.ref_dem != "")
    dem_file = opt.ref_dem;
  if (dem_file != "") {
    std::string file_type = asp::get_cloud_type(dem_file);
    if (file_type == "DEM") {
      vw::cartography::GeoReference georef;
      bool is_good = vw::cartography::read_georeference(georef, dem_file);
      if (!is_good)
        vw_throw( ArgumentErr() << "The DEM " << dem_file
                  << " does not have a georeference.\n");

      if (opt.datum_str == "" ) {
        opt.datum = georef.datum();
        opt.datum_str = opt.datum.name();
        guessed_datum = true;
      }
    }
  }
  
  // Set the datum, either based on what the user specified or the axes
  if (opt.datum_str != "" && !guessed_datum) {
    try {
      opt.datum.set_well_known_datum(opt.datum_str);
    } catch(...) {
      // Whatever datum name we had, it was bad, so we'll make more attempts below
      opt.datum_str = "";
      guessed_datum = false;
    }
  }else if (opt.semi_major > 0 && opt.semi_minor > 0){
    // Otherwise, if the user set the semi-axes, use that.
    opt.datum = cartography::Datum("User Specified Datum",
                                   "User Specified Spheroid",
                                   "Reference Meridian",
                                   opt.semi_major, opt.semi_minor, 0.0);
    opt.datum_str = opt.datum.name();
    guessed_datum = true;
  }

  // Otherwise try to set the datum based on cameras.  It will return
  // WGS84 if all else fails.
  // TODO(oalexan1): That may not be desirable with ground-level cameras.
  if (opt.datum_str == "") {
    asp::datum_from_cameras(opt.image_files, opt.camera_files,  
                            opt.stereo_session,  // may change
                            // Outputs
                            opt.datum);
    opt.datum_str = opt.datum.name();
  }
  
  // Many times the datum is mandatory
  if (opt.datum_str == "") {
    if (!opt.gcp_files.empty() || !opt.camera_position_file.empty() )
      vw_throw( ArgumentErr() << "When ground control points or a camera position file are used, "
                << "the datum must be specified.\n");
    
    if (opt.elevation_limit[0] < opt.elevation_limit[1])
      vw_throw( ArgumentErr()
                << "When filtering by elevation limit, the datum must be specified.\n");
  }

  vw_out() << "Will use the datum:\n" << opt.datum << std::endl;

  // This is a little clumsy, but need to see whether the user set --max-iterations
  // or --num-iterations. They are aliases to each other.
  if (!vm["max-iterations"].defaulted() && !vm["num-iterations"].defaulted()) 
    vw_throw( ArgumentErr() << "Cannot set both --num-iterations and --max-iterations.\n");
  if (!vm["max-iterations"].defaulted())
    opt.num_iterations = max_iterations_tmp;
  
  if ( opt.out_prefix.empty() )
    vw_throw( ArgumentErr() << "Missing output prefix.\n" << usage << general_options  );

  // Create the output directory
  vw::create_out_dir(opt.out_prefix);

  // Turn on logging to file
  asp::log_to_file(argc, argv, "", opt.out_prefix);

  opt.load_intrinsics_options(intrinsics_to_float_str, intrinsics_to_share_str,
                              !vm["intrinsics-to-share"].defaulted());

  opt.parse_intrinsics_limits(intrinsics_limit_str);

  boost::to_lower(opt.cost_function);

  if (opt.apply_initial_transform_only && opt.initial_transform_file == "")
    vw_throw(vw::IOErr() << "Cannot use --apply-initial-transform-only "
              << "without --initial-transform.\n");
  
  if (opt.initial_transform_file != "") {
    vw_out() << "Reading the alignment transform from: " << opt.initial_transform_file << "\n";
    vw::read_matrix_as_txt(opt.initial_transform_file, opt.initial_transform);
    if (opt.initial_transform.cols() != 4 || opt.initial_transform.rows() != 4)
      vw_throw(ArgumentErr() << "Could not read the initial transform.\n");
    vw_out() << "Initial transform:\n" << opt.initial_transform << std::endl;
  }

  // Parse the indices of cameras not to float
  if (opt.fixed_cameras_indices_str != "") {
    opt.fixed_cameras_indices.clear();
    std::istringstream is(opt.fixed_cameras_indices_str);
    int val;
    while (is >> val) {
      opt.fixed_cameras_indices.insert(val);
      if (val < 0 || val >= (int)opt.image_files.size()) 
        vw_throw( vw::IOErr() << "The camera index to keep fixed " << val
                              << " is out of bounds.\n" );
    }
  }

  if (!opt.fixed_cameras_indices.empty() && !opt.fixed_image_list.empty())
    vw_throw(ArgumentErr() << "Cannot specify both --fixed-camera-indices and "
             << "--fixed-image-list.\n");
  if (!opt.fixed_image_list.empty()) {

    opt.fixed_cameras_indices.clear();
    
    std::vector<std::string> fixed_images;
    asp::read_list(opt.fixed_image_list, fixed_images);

    // Find the indices of all images
    std::map<std::string, int> all_indices;
    for (size_t image_it = 0; image_it < opt.image_files.size(); image_it++) 
      all_indices[opt.image_files[image_it]] = image_it;

    // Find the indices of images to fix
    for (size_t image_it = 0; image_it < fixed_images.size(); image_it++) {
      auto map_it = all_indices.find(fixed_images[image_it]);
      if (map_it == all_indices.end())
        vw_throw(ArgumentErr() << "Could not find image " << fixed_images[image_it]
                 << " read via --fixed-image-list among the input images.\n");
      opt.fixed_cameras_indices.insert(map_it->second);
    }
  }
  
  if (opt.reference_terrain != "") {
    std::string file_type = asp::get_cloud_type(opt.reference_terrain);
    if (file_type == "CSV" && opt.csv_format_str == "") 
      vw_throw( ArgumentErr() << "When using a csv reference terrain, "
                              << "must specify the csv-format.\n");
    if (opt.datum_str == "")
      vw_throw( ArgumentErr() << "When using a reference terrain, must specify the datum.\n");
    if (opt.disparity_list == "") 
      vw_throw( ArgumentErr() << "When using a reference terrain, must specify a list "
                              << "of disparities.\n");
    if (opt.max_disp_error <= 0) 
      vw_throw( ArgumentErr() << "Must specify --max-disp-error in pixels as a positive value.\n");
    if (opt.reference_terrain_weight < 0) 
      vw_throw( ArgumentErr() << "The value of --reference-terrain-weight must be non-negative.\n");
  }

  if (opt.match_files_prefix != "" && opt.clean_match_files_prefix != "") 
    vw_throw( ArgumentErr()
              << "Cannot specify both --match-files-prefix and --clean-match-files-prefix.\n");

  if (int(opt.proj_win != BBox2(0, 0, 0, 0)) + int(!opt.proj_str.empty()) == 1)
    vw_throw(ArgumentErr() << "Must specify both or neither of --proj-win and --proj-str.\n");

  if (int(opt.transform_cameras_using_gcp) +
      int(opt.transform_cameras_with_shared_gcp) +
      int(opt.init_camera_using_gcp) > 1)
    vw::vw_throw(vw::ArgumentErr()
                 << "Cannot specify more than one of --transform-cameras-using-gcp, "
                 << "transform-cameras-with-shared-gcp, init-camera-using-gcp.\n");

  return;
}

// A wrapper around ip matching. Can also work with NULL cameras.
void ba_match_ip(Options & opt, SessionPtr session, 
                 std::string const& image1_path,  std::string const& image2_path,
                 vw::camera::CameraModel* cam1,   vw::camera::CameraModel* cam2,
                 std::string const& match_filename) {
  
  boost::shared_ptr<DiskImageResource>
    rsrc1(vw::DiskImageResourcePtr(image1_path)),
    rsrc2(vw::DiskImageResourcePtr(image2_path));
  if ( (rsrc1->channels() > 1) || (rsrc2->channels() > 1) )
    vw_throw(ArgumentErr()
             << "Error: Input images can only have a single channel!\n\n");
  float nodata1, nodata2;
  asp::get_nodata_values(rsrc1, rsrc2, nodata1, nodata2);

  // IP matching may not succeed for all pairs
  
  // Get masked views of the images to get statistics from
  DiskImageView<float> image1_view(rsrc1), image2_view(rsrc2);
  ImageViewRef< PixelMask<float> > masked_image1
    = create_mask_less_or_equal(image1_view,  nodata1);
  ImageViewRef< PixelMask<float> > masked_image2
    = create_mask_less_or_equal(image2_view, nodata2);
  
  // Since we computed statistics earlier, this will just be loading files.
  vw::Vector<vw::float32,6> image1_stats, image2_stats;
  image1_stats = asp::gather_stats(masked_image1, image1_path, 
                                   opt.out_prefix, image1_path);
  image2_stats = asp::gather_stats(masked_image2, image2_path, 
                                   opt.out_prefix, image2_path);
  
  // Do not save by default .vwip files as those take space and are
  // not needed after a match file is created. If the user wants them,
  // they must be saved in a subdirectory for each match pair, as
  // .vwip files change depending on the pair.
  std::string ip_file1 = "", ip_file2 = "";
  if (opt.save_vwip) {
      // parallel_bundle_adjust should have set vwip_prefix, but not bundle_adjust itself
    if (opt.vwip_prefix == "")
      opt.vwip_prefix = opt.out_prefix; 
    
    ip_file1 = ip::ip_filename(opt.vwip_prefix, image1_path); 
    ip_file2 = ip::ip_filename(opt.vwip_prefix, image2_path);
    vw::create_out_dir(opt.vwip_prefix);
  }
  
  // The match files (.match) are cached unless the images or camera
  // are newer than them.
  session->ip_matching(image1_path, image2_path,
                       Vector2(masked_image1.cols(), masked_image1.rows()),
                       image1_stats, image2_stats, 
                       nodata1, nodata2, cam1, cam2, match_filename, 
                       ip_file1, ip_file2);
}

//==================================================================================
// Mapprojected image functions.

/// If the user map-projected the images (this is useful when the
/// perspective or illumination conditions are too different, and
/// automated matching fails), first create matches among the
/// mapprojected images (or use any such matches created beforehand
/// manually by the user), and project those matches into the cameras,
/// creating matches between the raw images that then bundle_adjust
/// can use. Both matches between mapprojected images and between
/// original images are saved to files.
void matches_from_mapproj_images(int i, int j,
                                 Options& opt, SessionPtr session,
                                 std::vector<std::string> const& map_files,
                                 vw::cartography::GeoReference const& dem_georef,
                                 ImageViewRef<PixelMask<double>> & interp_dem,
                                 std::string const& match_filename){
  
  vw::cartography::GeoReference georef1, georef2;
  vw_out() << "Reading georef from " << map_files[i] << ' ' << map_files[j] << std::endl;
  bool is_good1 = vw::cartography::read_georeference(georef1, map_files[i]);
  bool is_good2 = vw::cartography::read_georeference(georef2, map_files[j]);
  if (!is_good1 || !is_good2) {
    vw_throw(ArgumentErr() << "Error: Cannot read georeference.\n");
  }
  
  std::string image1_path  = opt.image_files[i];
  std::string image2_path  = opt.image_files[j];
  if (boost::filesystem::exists(match_filename)) {
    vw_out() << "Using cached match file: " << match_filename << "\n";
    return;
  }

  if (opt.skip_matching)
    return;

  // If the match file does not exist, create it. The user can create this manually
  // too. 
  std::string map_match_file = ip::match_filename(opt.out_prefix,
                                                  map_files[i], map_files[j]);
  try{
    
    ba_match_ip(opt, session, map_files[i], map_files[j],
                NULL, NULL, // cameras are set to null since images are mapprojected
                map_match_file);
  } catch ( const std::exception& e ){
    vw_out() << "Could not find interest points between images "
             << map_files[i] << " and " << map_files[j] << std::endl;
    vw_out(WarningMessage) << e.what() << std::endl;
    return;
  } //End try/catch
  
  if (!boost::filesystem::exists(map_match_file)) {
    vw_out() << "Missing: " << map_match_file << "\n";
    return;
  }

  vw_out() << "Reading: " << map_match_file << std::endl;
  std::vector<ip::InterestPoint> ip1,     ip2;
  std::vector<ip::InterestPoint> ip1_cam, ip2_cam;
  ip::read_binary_match_file(map_match_file, ip1, ip2);
  
  // Undo the map-projection
  for (size_t ip_iter = 0; ip_iter < ip1.size(); ip_iter++) {
    
    vw::ip::InterestPoint P1 = ip1[ip_iter];
    vw::ip::InterestPoint P2 = ip2[ip_iter];
    if (!asp::projected_ip_to_raw_ip(P1, interp_dem, opt.camera_models[i], georef1, dem_georef))
      continue;
    if (!asp::projected_ip_to_raw_ip(P2, interp_dem, opt.camera_models[j], georef2, dem_georef))
      continue;
    
    ip1_cam.push_back(P1);
    ip2_cam.push_back(P2);
  }
  
  vw_out() << "Saving " << ip1_cam.size() << " matches.\n";
  
  vw_out() << "Writing: " << match_filename << std::endl;
  ip::write_binary_match_file(match_filename, ip1_cam, ip2_cam);

} // End function matches_from_mapproj_images()

/// If the user map-projected the images and created matches by hand
/// from each map-projected image to the DEM it was map-projected onto,
/// project those matches back into the camera image, and create gcp
/// tying each camera image match to its desired location on the DEM.
void create_gcp_from_mapprojected_images(Options const& opt){

  // Read the map-projected images and the dem
  std::istringstream is(opt.gcp_from_mapprojected);
  std::vector<std::string> image_files;
  std::string file;
  while (is >> file){
    image_files.push_back(file); 
  }
  std::string dem_file = image_files.back();
  image_files.erase(image_files.end() - 1); // wipe the dem from the list

  vw::cartography::GeoReference dem_georef;
  ImageViewRef<PixelMask<double>> interp_dem;
  asp::create_interp_dem(dem_file, dem_georef, interp_dem);

  int num_images = image_files.size();
  std::vector<std::vector<vw::ip::InterestPoint> > matches;
  std::vector<vw::cartography::GeoReference> img_georefs;
  matches.resize(num_images + 1); // the last match will be for the DEM

  // Read the matches and georefs
  for (int i = 0; i < num_images; i++) {

    vw::cartography::GeoReference img_georef;
    vw_out() << "Reading georef from " << image_files[i]  << std::endl;
    bool is_good_img = vw::cartography::read_georeference(img_georef, image_files[i]);
    if (!is_good_img) {
      vw_throw(ArgumentErr() << "Error: Cannot read georeference.\n");
    }
    img_georefs.push_back(img_georef);

    std::string match_filename = ip::match_filename(opt.out_prefix,
                                                    image_files[i], dem_file);
    if (!boost::filesystem::exists(match_filename)) 
      vw_throw(ArgumentErr() << "Missing: " << match_filename << ".\n");

    vw_out() << "Reading: " << match_filename << std::endl;
    std::vector<ip::InterestPoint> ip1, ip2;
    ip::read_binary_match_file(match_filename, ip1, ip2);

    if (matches[num_images].size() > 0 && matches[num_images].size() != ip2.size()) {
      vw_throw(ArgumentErr() << "All match files must have the same number of IP.\n");
    }
    matches[i]          = ip1;
    matches[num_images] = ip2;
  }

  std::vector<std::vector<vw::ip::InterestPoint> > cam_matches = matches;

  std::string gcp_file;
  for (int i = 0; i < num_images; i++) {
    gcp_file += boost::filesystem::basename(opt.image_files[i]);
    if (i < num_images - 1) gcp_file += "__"; 
  }
  gcp_file = opt.out_prefix + "-" + gcp_file + ".gcp";

  vw_out() << "Writing: " << gcp_file << std::endl;
  std::ofstream output_handle(gcp_file.c_str());
  output_handle.precision(17);
  
  int num_ips = matches[0].size();
  int pts_count = 0;
  for (int p = 0; p < num_ips; p++) { // Loop through IPs

    // Compute the GDC coordinate of the point
    ip::InterestPoint dem_ip = matches[num_images][p];
    Vector2 dem_pixel(dem_ip.x, dem_ip.y);
    Vector2 lonlat = dem_georef.pixel_to_lonlat(dem_pixel);

    if (!interp_dem.pixel_in_bounds(dem_pixel)) {
      vw_out() << "Skipping pixel outside of DEM: " << dem_pixel << std::endl;
      continue;
    }

    PixelMask<float> mask_height = interp_dem(dem_pixel[0], dem_pixel[1])[0];
    if (!is_valid(mask_height)) continue;

    Vector3 llh(lonlat[0], lonlat[1], mask_height.child());
    //Vector3 dem_xyz = dem_georef.datum().geodetic_to_cartesian(llh);

    // The ground control point ID
    output_handle << pts_count;
    // Lat, lon, height
    output_handle << ", " << lonlat[1] << ", " << lonlat[0] << ", " << mask_height.child();
    // Sigma values
    output_handle << ", " << 1 << ", " << 1 << ", " << 1;

    // Write the per-image information
    for (int i = 0; i < num_images; i++) {

      // Take the ip in the map-projected image, and back-project it into the camera
      ip::InterestPoint ip = matches[i][p];
      if (!asp::projected_ip_to_raw_ip(ip, interp_dem, opt.camera_models[i],
                                  img_georefs[i], dem_georef))
          continue;

      // TODO: Here we can have a book-keeping problem!
      cam_matches[i][p] = ip;

      output_handle << ", " << opt.image_files[i];
      output_handle << ", " << ip.x << ", " << ip.y; // IP location in image
      output_handle << ", " << 1 << ", " << 1; // Sigma values
    } // End loop through IP sets
    output_handle << std::endl; // Finish the line
    pts_count++;

  } // End loop through IPs
  output_handle.close();

  // Write out match files for each pair of images.
  for (int i = 0; i < num_images; i++) {
    for (int j = i+1; j < num_images; j++) {
      std::string image1_path    = opt.image_files[i];
      std::string image2_path    = opt.image_files[j];
      std::string match_filename = ip::match_filename(opt.out_prefix, image1_path, image2_path);

      vw_out() << "Writing: " << match_filename << std::endl;
      ip::write_binary_match_file(match_filename, cam_matches[i], cam_matches[j]);
    }
  }

}

// End map projection functions

int main(int argc, char* argv[]) {

  Options opt;
  try {
    xercesc::XMLPlatformUtils::Initialize();

    handle_arguments(argc, argv, opt);

    asp::load_cameras(opt.image_files, opt.camera_files, opt.out_prefix, opt,  
                      opt.approximate_pinhole_intrinsics,  
                      // Outputs
                      opt.stereo_session,  // may change
                      opt.single_threaded_cameras,  
                      opt.camera_models);
    
    // Prepare for computing footprints of images
    std::string dem_file_for_overlap;
    double pct_for_overlap = -1.0;
    if (opt.auto_overlap_params != "") {
      std::istringstream is(opt.auto_overlap_params);
      if (!(is >> dem_file_for_overlap >> pct_for_overlap)) 
        vw_throw(ArgumentErr() << "Could not parse correctly option --auto-overlap-params.\n");
    }

    // For when we make matches based on mapprojected images. Read mapprojected
    // images and a DEM from either command line or a list.
    std::vector<std::string> map_files;
    vw::cartography::GeoReference dem_georef;
    ImageViewRef<PixelMask<double>> interp_dem;
    if (!opt.apply_initial_transform_only) {
      
      if (!opt.mapprojected_data_list.empty()) {
        asp::read_list(opt.mapprojected_data_list, map_files);
        opt.mapprojected_data = "non-empty"; // put a token value, to make it non-empty
      } else if (opt.mapprojected_data != "") {
        std::istringstream is(opt.mapprojected_data);
        std::string file;
        while (is >> file)
          map_files.push_back(file); 
      }

      if (!opt.mapprojected_data.empty()) {
        if (opt.camera_models.size() + 1 != map_files.size()) 
          vw_throw(ArgumentErr() << "Error: Expecting as many mapprojected images as "
                   << "cameras, and also a DEM.\n");
        
        std::string dem_file = map_files.back();
        map_files.erase(map_files.end() - 1);
        
        asp::create_interp_dem(dem_file, dem_georef, interp_dem);
      }
    }
    
    // Assign the images which this instance should compute statistics for.
    std::vector<size_t> image_stats_indices;
    int num_images = opt.image_files.size();

    for (size_t i = opt.instance_index; i < num_images; i += opt.instance_count)
      image_stats_indices.push_back(i);

    // Compute statistics for the designated images (or mapprojected
    // images), and perhaps the footprints
    // TODO(oalexan1): Make this into a function
    for (size_t i = 0; i < image_stats_indices.size(); i++) {

      if (opt.apply_initial_transform_only)
        continue; // no stats need to happen

      if (opt.skip_matching || opt.clean_match_files_prefix != "" ||
          opt.match_files_prefix != "")
        continue;
      
      size_t index = image_stats_indices[i];

      // The stats need to be computed for the mapprojected image, if provided
      std::string image_path;
      if (map_files.empty()) 
        image_path = opt.image_files[index];
      else
        image_path = map_files[index];
      
      // Call a bunch of stuff to get the nodata value
      boost::shared_ptr<DiskImageResource> rsrc(vw::DiskImageResourcePtr(image_path));
      float nodata, dummy;
      asp::get_nodata_values(rsrc, rsrc, nodata, dummy);

      // Set up the image view
      DiskImageView<float> image_view(rsrc);
      ImageViewRef< PixelMask<float> > masked_image
        = create_mask_less_or_equal(image_view,  nodata);

      // Use caching function call to compute the image statistics.
      asp::gather_stats(masked_image, image_path, opt.out_prefix, image_path);

      // Compute and cache the camera footprint bbox
      if (opt.auto_overlap_params != "")
        asp::camera_bbox_with_cache(dem_file_for_overlap,
                                    opt.image_files[index], // use the original image
                                    opt.camera_models[index],  
                                    opt.out_prefix);
    }
    
    // Done computing image statistics.

    if (opt.stop_after_stats) {
      vw_out() << "Quitting after statistics computation.\n";
      xercesc::XMLPlatformUtils::Terminate();

      return 0;
    }

    // Calculate which images overlap
    if (opt.auto_overlap_params != "") {
      opt.have_overlap_list = true;
      asp::build_overlap_list_based_on_dem(opt.out_prefix,  
                                           dem_file_for_overlap, pct_for_overlap,
                                           opt.image_files, opt.camera_models,
                                           // output
                                           opt.overlap_list);
    }

    // Create the match points. Iterate through each pair of input images.

    // Load estimated camera positions if they were provided.
    std::vector<Vector3> estimated_camera_gcc;
    load_estimated_camera_positions(opt, estimated_camera_gcc);
    const bool got_est_cam_positions =
      (estimated_camera_gcc.size() == static_cast<size_t>(num_images));

    // Find interest points between all of the image pairs.
    
    // Make a list of all the image pairs to find matches for
    std::vector<std::pair<int,int> > all_pairs;
    if (!opt.apply_initial_transform_only)
      asp::determine_image_pairs(// Inputs
                                 opt.overlap_limit, opt.match_first_to_last,  
                                 opt.image_files, 
                                 got_est_cam_positions, opt.position_filter_dist,
                                 estimated_camera_gcc,
                                 opt.have_overlap_list,
                                 opt.overlap_list,
                                 // Output
                                 all_pairs);

    // Create GCP from mapprojection
    if (opt.gcp_from_mapprojected != "" && !opt.apply_initial_transform_only) {
      create_gcp_from_mapprojected_images(opt);
      return 0;
    }

    // TODO: Make this a function
    // Assign the matches which this instance should compute.
    // This is for when called from parallel_bundle_adjust.
    size_t per_instance = all_pairs.size() / opt.instance_count; // Round down
    size_t remainder    = all_pairs.size() % opt.instance_count;
    size_t start_index  = 0, this_count = 0;
    for (size_t i = 0; i <= opt.instance_index; i++) {
      this_count = per_instance;
      if (i < remainder)
        ++this_count;
      start_index += this_count;
    }
    start_index -= this_count;

    // TODO(oalexan1): The above logic is confusing. It is some
    // kind of partitioning. At least when parallel_bundle_adjust
    // is not invoked, for now check that things are as expected,
    // so all the matches are used.
    if (opt.instance_count == 1) {
      if (start_index != 0 || this_count != all_pairs.size()) 
        vw::vw_throw(vw::ArgumentErr() << "Book-keeping failure in bundle_adjust.\n");
    }
    
    std::vector<std::pair<int,int>> this_instance_pairs;
    for (size_t i=0; i<this_count; i++)
      this_instance_pairs.push_back(all_pairs[i+start_index]);

    // When using match-files-prefix or 
    // clean_match_files_prefix, form the list of match files, rather
    // than searching for them exhaustively on disk, which can get
    // very slow.
    bool external_matches = (!opt.clean_match_files_prefix.empty() ||
                             !opt.match_files_prefix.empty());
    std::set<std::string> existing_files;
    if (external_matches) {
      std::string prefix = asp::match_file_prefix(opt.clean_match_files_prefix,
                                                  opt.match_files_prefix,  
                                                  opt.out_prefix);
      vw_out() << "Computing the list of existing match files.\n";
      asp::listExistingMatchFiles(prefix, existing_files);
    }
    
    // Process the selected pairs
    for (size_t k = 0; k < this_instance_pairs.size(); k++) {

      if (opt.apply_initial_transform_only)
        continue;
      
      const int i = this_instance_pairs[k].first;
      const int j = this_instance_pairs[k].second;

      std::string const& image1_path  = opt.image_files[i];  // alias
      std::string const& image2_path  = opt.image_files[j];  // alias
      std::string const& camera1_path = opt.camera_files[i]; // alias
      std::string const& camera2_path = opt.camera_files[j]; // alias
      
      // See if perhaps to load match files from a different source
      std::string match_file 
        = asp::match_filename(opt.clean_match_files_prefix, opt.match_files_prefix,  
                              opt.out_prefix, image1_path, image2_path);

      // The external match file does not exist, don't try to load it
      if (external_matches && existing_files.find(match_file) == existing_files.end())
        continue;
     
      opt.match_files[std::make_pair(i, j)] = match_file;

      // If we skip matching (which is the case, among other situations, when
      // using external matches), there's no point in checking if the match
      // files are recent.
      bool inputs_changed = false;
      if (!opt.skip_matching) {
        inputs_changed = (!asp::is_latest_timestamp(match_file,
                                                    image1_path,  image2_path,
                                                    camera1_path, camera2_path));

        // We make an exception and not rebuild if explicitly asked
        if (asp::stereo_settings().force_reuse_match_files &&
            boost::filesystem::exists(match_file))
          inputs_changed = false;
      }
      
      if (!inputs_changed) {
        vw_out() << "\t--> Using cached match file: " << match_file << "\n";
        continue;
      }

      // Read no-data
      boost::shared_ptr<DiskImageResource>
        rsrc1(vw::DiskImageResourcePtr(image1_path)),
        rsrc2(vw::DiskImageResourcePtr(image2_path));
      if ((rsrc1->channels() > 1) || (rsrc2->channels() > 1))
        vw_throw(ArgumentErr() << "Error: Input images can only have a single channel!\n\n");
      float nodata1, nodata2;
      asp::get_nodata_values(rsrc1, rsrc2, nodata1, nodata2);
      
      // Set up the stereo session
      SessionPtr session(asp::StereoSessionFactory::create(opt.stereo_session, // may change
                                                           opt, image1_path,  image2_path,
                                                           camera1_path, camera2_path,
                                                           opt.out_prefix));


      // Find matches between image pairs. This may not always succeed.
      try{

        if (opt.mapprojected_data == "") 
          ba_match_ip(opt, session, image1_path, image2_path,
                      opt.camera_models[i].get(),
                      opt.camera_models[j].get(),
                      match_file);

        else
          matches_from_mapproj_images(i, j, opt, session, map_files, dem_georef, interp_dem,  
                                      match_file);

        // Compute the coverage fraction
        std::vector<ip::InterestPoint> ip1, ip2;
        ip::read_binary_match_file(match_file, ip1, ip2);
        int right_ip_width = rsrc1->cols() *
                              static_cast<double>(100-opt.ip_edge_buffer_percent)/100.0;
        Vector2i ip_size(right_ip_width, rsrc1->rows());
        double ip_coverage = asp::calc_ip_coverage_fraction(ip2, ip_size);
        vw_out() << "IP coverage fraction = " << ip_coverage << std::endl;
      } catch (const std::exception& e){
        vw_out() << "Could not find interest points between images "
                  << opt.image_files[i] << " and " << opt.image_files[j] << std::endl;
        vw_out(WarningMessage) << e.what() << std::endl;
      } //End try/catch
    } // End loop through all input image pairs

    if (opt.stop_after_matching){
      vw_out() << "Quitting after matches computation.\n";
      return 0;
    }

    // All the work happens here! It also writes out the results.
    do_ba_ceres(opt, estimated_camera_gcc);

    xercesc::XMLPlatformUtils::Terminate();

  } ASP_STANDARD_CATCHES;
}
