#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file SpiceUtilities.cc
///

#include <asp/SpiceIO/SpiceUtilities.h>

#include "SpiceUsr.h"
#include "SpiceZfc.h"

#include <iostream>
#include <fstream>
#include <list>
#include <string>

#include <vw/Core/Exception.h>
#include <vw/Core/Debugging.h>

#include <string.h>

using namespace std;
using namespace vw;

namespace asp {
namespace spice {

  // The maximum SPICE message length (currently) is 23*80 = 1840
  // chars. This is not definied anywhere so we define it here.

  enum { LONG_MSG_LEN = 1840 };

  void CHECK_SPICE_ERROR() {
    char longms[LONG_MSG_LEN + 1];

    longms[LONG_MSG_LEN] = 0;                // ensure the string is terminated
    if (failed_c()) {
      getlms_(longms, LONG_MSG_LEN);
      reset_c();
      // Trim off the white space at the end
      for (int i = LONG_MSG_LEN-1; (i >= 0) && (longms[i] == ' '); --i)
        longms[i] = 0;
      cout
        << "SPICE: An error occurred when accessing the SPICE information:\n\n"
        << longms;
      throw spice::SpiceErr()
        << "SPICE: An error occurred when accessing the SPICE information:\n\n"
        << longms;
    }
  }

  // Convert from SCLK to ephemeris time for a spacecraft with a given
  // NAIF ID.
  double sclk_to_et(std::string sclk, int naif_id) {
    SpiceDouble query_time;

    scs2e_c( naif_id, sclk.c_str(), &query_time );
    CHECK_SPICE_ERROR();
    return query_time;
  }

  // Convert from Ephemeris time to UTC time.
  string et_to_utc(double ephemeris_time) {
    char utc_time[18];
    timout_c( ephemeris_time, "MM/DD/YYYY HR:MN:SC", 20, utc_time );
    CHECK_SPICE_ERROR();
    return string(utc_time);
  }

  // Convert from Ephemeris time to UTC time.
  double utc_to_et(std::string const& utc) {
    SpiceDouble et;
    std::string tmp = utc;
    // Spice can't cope with the optional trailing 'Z'
    if( tmp[tmp.size()-1] == 'Z' ) tmp.erase(tmp.size()-1,1);
    utc2et_c( tmp.c_str(), &et );
    CHECK_SPICE_ERROR();
    return et;
  }

  void kernel_param(std::string const& key, double &value) {
    SpiceInt nvalues;
    SpiceBoolean found;
    gdpool_c(key.c_str(), 0, 1, &nvalues, &value, &found);
    if (!found) { vw_throw(LogicErr() << "spice::kernel_param() could not find value for key \"" << key << "\""); }
  }

  template <int ElemN>
  void kernel_param(std::string const& key, Vector<double, ElemN> &value) {
    SpiceDouble elements[ElemN];
    SpiceInt nvalues;
    SpiceBoolean found;
    gdpool_c(key.c_str(), 0, ElemN, &nvalues, elements, &found);
    if (!found) { vw_throw(LogicErr() << "spice::kernel_param() could not find value for key \"" << key << "\""); }
    if (ElemN != nvalues) { vw_throw(LogicErr() << "spice::kernel_param() returned fewer elements than requested"); }
    for (int i=0; i < ElemN; ++i)
      value[i] = elements[i];
  }

  // Explicit instantiation
  template void kernel_param<2>(std::string const& key, Vector<double, 2> &value);
  template void kernel_param<3>(std::string const& key, Vector<double, 3> &value);
  template void kernel_param<4>(std::string const& key, Vector<double, 4> &value);

  // Load the state of a camera for a given time.
  void body_state(double time_,
                  Vector3 &position,
                  Vector3 &velocity,
                  Quat &pose,
                  std::string const& spacecraft,
                  std::string const& reference_frame,
                  std::string const& planet,
                  std::string const& instrument) {

    SpiceDouble time = time_;

    // Obtain the state vector of the spacecraft at the given
    // ephemeris time.
    //
    // The position and velocity of the spacecraft will be reported
    // relative to the MARS IAU2000 coordinate frame.  No light time
    // correction is made here since the spacecraft and the planet
    // surface (where the light originated) are very close.  (Normally
    // light time correction is computed between the origins of the
    // two coordinate frames.)
    SpiceDouble state[6];
    SpiceDouble light_time;

    spkezr_c( spacecraft.c_str(),  time,  reference_frame.c_str(), "NONE", planet.c_str(), state, &light_time );

    // Output is in km and km/s, so we must convert the state array to
    // units of m and m/s.
    position(0) = state[0] * 1000.0;
    position(1) = state[1] * 1000.0;
    position(2) = state[2] * 1000.0;
    velocity(0) = state[3] * 1000.0;
    velocity(1) = state[4] * 1000.0;
    velocity(2) = state[5] * 1000.0;

    // Get pose data from the spacecraft CK kernel
    //
    // Here, we explicity get the pose of the camera relative to the
    // Mars frame.  This incorporates the additional, slight rotation
    // from the spacecraft frame to the camera frame.
    SpiceDouble rotation_matrix[3][3];
    pxform_c( reference_frame.c_str(), instrument.c_str(), time, rotation_matrix);

    // Convert that matrix into a quaternion.
    SpiceDouble quaternion[4];
    m2q_c( rotation_matrix, quaternion );
    pose = Quat(quaternion[0],quaternion[1],quaternion[2],quaternion[3]);

    CHECK_SPICE_ERROR();
  }


  // Load the state of the MOC camera for a given time range, returning
  // observations of the state for the given time interval.
  void body_state(double begin_time, double end_time, double interval,
                  std::vector<Vector3> &position,
                  std::vector<Vector3> &velocity,
                  std::vector<Quat > &pose,
                  std::string const& spacecraft,
                  std::string const& reference_frame,
                  std::string const& planet,
                  std::string const& instrument) {

    unsigned int number_of_samples = (unsigned int)ceil((end_time - begin_time) / interval);

    position.resize(number_of_samples);
    velocity.resize(number_of_samples);
    pose.resize(number_of_samples);

    unsigned int interval_count = 0;
    for( SpiceDouble time = begin_time;
         time < end_time;
         time += interval ) {
      // Obtain the state vector of the spacecraft at the given
      // ephemeris time.
      //
      // The position and velocity of the spacecraft will be reported
      // relative to the MARS IAU2000 coordinate frame.  No light time
      // correction is made here since the spacecraft and the planet
      // surface (where the light originated) are very close.  (Normally
      // light time correction is computed between the origins of the
      // two coordinate frames.)
      SpiceDouble state[6];
      SpiceDouble light_time;

      spkezr_c( spacecraft.c_str(),  time,  reference_frame.c_str(), "NONE", planet.c_str(), state, &light_time );

      // Output is in km and km/s, so we must convert the state array to
      // units of m and m/s.
      position[interval_count](0) = state[0] * 1000.0;
      position[interval_count](1) = state[1] * 1000.0;
      position[interval_count](2) = state[2] * 1000.0;
      velocity[interval_count](0) = state[3] * 1000.0;
      velocity[interval_count](1) = state[4] * 1000.0;
      velocity[interval_count](2) = state[5] * 1000.0;

      // Get pose data from the spacecraft CK kernel
      //
      // Here, we explicity get the pose of the camera relative to the
      // Mars frame.  This incorporates the additional, slight rotation
      // from the spacecraft frame to the camera frame.
      SpiceDouble rotation_matrix[3][3];
      pxform_c( reference_frame.c_str(), instrument.c_str(), time, rotation_matrix);

      // Convert that matrix into a quaternion.
      SpiceDouble quaternion[4];
      m2q_c( rotation_matrix, quaternion );
      pose[interval_count] = Quat(quaternion[0],quaternion[1],quaternion[2],quaternion[3]);
      interval_count++;
    }

    CHECK_SPICE_ERROR();
  }

  // Load all relevent SPICE kernels.
  //
  // Someday, rather than hard coding these values, the user might be
  // able to specify which kernel files to load in the stereo.default
  // file or a similar configuration file.
  void load_kernels(std::list<std::string> &kernels) {

    // Alter the spice error handling behavior to allow us to handle
    // errors ourselves.
    int lenout = 0;
    char set[] = "SET", ret[] = "RETURN";
    erract_c (  set, lenout, ret  );

    // Load the kernels
    list<string>::iterator iter;
    for (iter = kernels.begin(); iter != kernels.end(); iter++)
      furnsh_c( (*iter).c_str() );

    CHECK_SPICE_ERROR();
  }

  // This variant reads a file containing a list of kernel files,
  // one per line.  The optional second parameter specifies a
  // path prefix.
  void load_kernels(std::string const& kernels_file, std::string const& prefix) {
    std::ifstream input_file;
    input_file.open(kernels_file.c_str(), std::ios::in);
    if (!input_file.good())
      vw_throw(IOErr() << "An error occurred while opening the kernels file for reading.");

    std::list<std::string> kernel_list;
    char line[1024];
    while (!input_file.eof()) {
      input_file.getline(line,  1024);
      if (strlen(line) > 0) {
        kernel_list.push_back(prefix+line);
        vw_out(DebugMessage) << "Adding kernel....... \"" << (prefix+line) << "\"." << std::endl;;
      }
    }

    load_kernels(kernel_list);
  }

}} // namespace asp::spice
