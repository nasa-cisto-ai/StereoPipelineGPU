// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file StereoSessionFactory.cc
///

// This include must exist for linking purposes
#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/Sessions/StereoSessionMapProj.h>
#include <asp/Sessions/StereoSessionIsis.h>
#include <asp/Sessions/StereoSessionNadirPinhole.h>
#include <asp/Sessions/StereoSessionPinhole.h>
#include <asp/Sessions/StereoSessionRPC.h>
#include <asp/Sessions/StereoSessionASTER.h>

#include <vw/FileIO/DiskImageResourceRaw.h>
#include <vw/Camera/CameraUtilities.h>
#include <asp/Camera/SPOT_XML.h>
#include <asp/Camera/ASTER_XML.h>
#include <vw/Camera/OpticalBarModel.h>

namespace asp {

  StereoSession* StereoSessionFactory::create(std::string      & session_type, // in-out
                                              vw::GdalWriteOptions const& options,
                                              std::string const& left_image_file,
                                              std::string const& right_image_file,
                                              std::string const& left_camera_file,
                                              std::string const& right_camera_file,
                                              std::string const& out_prefix,
                                              std::string const& input_dem,
                                              const bool allow_map_promote) {
    
    // Known user session types are:
    // DG, RPC, ISIS, Pinhole, NadirPinhole, OpticalBar
    //
    // Hidden sessions are:
    // DGMapRPC, Blank (Guessing)
    
    // Try to guess the session if not provided
    std::string actual_session_type = session_type;
    bool quiet = true;
    boost::to_lower(actual_session_type);
    if (actual_session_type.empty()) {
      if (asp::has_pinhole_extension(left_camera_file ) || // TODO: Fix this dangerous code!
          asp::has_pinhole_extension(right_camera_file)   ) {
        // There can be several types of .tsai files
        std::string error_pinhole, error_opticalbar;
        try {
          boost::shared_ptr<vw::camera::CameraModel> P = 
            vw::camera::load_pinhole_camera_model(left_camera_file);
          actual_session_type = "pinhole";
        }catch (std::exception & e) {
          error_pinhole = e.what();
          try {
            vw::camera::OpticalBarModel P;
            P.read(left_camera_file);
            actual_session_type = "opticalbar";
          }catch (std::exception & e) {
            error_opticalbar = e.what();
            vw_throw(vw::NoImplErr() << "Could not read the camera model " <<
                     left_camera_file << " as pinhole. "
                     << "The error was: " << error_pinhole << "\n"
                     << "Could not read it as opticalbar model either, the error was: "
                     << error_opticalbar);
          }
        }
      } else if (boost::iends_with(boost::to_lower_copy(left_camera_file ), ".json") ||
          boost::iends_with(boost::to_lower_copy(right_camera_file), ".json") ) {
        actual_session_type = "csm";
      }else if (boost::iends_with(boost::to_lower_copy(left_image_file  ), ".cub") ||
		boost::iends_with(boost::to_lower_copy(right_image_file ), ".cub") ||
		boost::iends_with(boost::to_lower_copy(left_camera_file ), ".cub") ||
		boost::iends_with(boost::to_lower_copy(right_camera_file), ".cub") ) {
        actual_session_type = "isis";
      } else if (boost::iends_with(boost::to_lower_copy(left_camera_file ), ".dim") ||
                 boost::iends_with(boost::to_lower_copy(right_camera_file), ".dim") ) {
        actual_session_type = "spot5";
      }else if (boost::iends_with(boost::to_lower_copy(left_camera_file ), ".xml") ||
		boost::iends_with(boost::to_lower_copy(right_camera_file), ".xml") ) {

        // Here we have several options for .xml files. Note that a
        // Digital Globe xml file has both linescan and RPC
        // models. The logic below favors the linescan sensor.
        if (actual_session_type.empty()) {
          
          // TODO(oalexan1): Try to peek in the xml file instead of doing this exhaustive
          // checking.
          
          // Try DG exact linescan model
          try {
            StereoSessionDG session;
            boost::shared_ptr<vw::camera::CameraModel>
              left_model  = session.camera_model(left_image_file,  left_camera_file, quiet),
              right_model = session.camera_model(right_image_file, right_camera_file, quiet);
            actual_session_type = "dg";
          } catch (...) {}
        }
        
        if (actual_session_type.empty()) {
          // Try PeruSat exact linescan model
          try {
            StereoSessionPeruSat session;
            boost::shared_ptr<vw::camera::CameraModel>
              left_model  = session.camera_model(left_image_file,  left_camera_file, quiet),
              right_model = session.camera_model(right_image_file, right_camera_file, quiet);
            actual_session_type = "perusat";
          } catch (...) {}
        }

        if (actual_session_type.empty()) {
          // Try Pleiades exact linescan model
          try {
            StereoSessionPleiades session;
            boost::shared_ptr<vw::camera::CameraModel>
              left_model  = session.camera_model(left_image_file,  left_camera_file, quiet),
              right_model = session.camera_model(right_image_file, right_camera_file, quiet);
            actual_session_type = "pleiades";
          } catch (...) {}
        }
        
      } // end considering the xml extension case

      // Try RPC, which can either have xml cameras or no cameras at all (if embedded
      // in the tif files).
      if (actual_session_type.empty()) {
        try {
          StereoSessionRPC session;
          boost::shared_ptr<vw::camera::CameraModel>
            left_model  = session.camera_model(left_image_file,  left_camera_file, quiet),
            right_model = session.camera_model(right_image_file, right_camera_file, quiet);
          actual_session_type = "rpc";
        } catch (...) {}
      }
    }
    
    if (allow_map_promote) {
      if (!input_dem.empty() && actual_session_type == "dg") {
        // User says DG but also gives a DEM.
        actual_session_type = "dgmaprpc";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: dgmaprpc.\n";
      }
      if (!input_dem.empty() && actual_session_type == "rpc") {
        // User says RPC but also gives a DEM.
        actual_session_type = "rpcmaprpc";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: rpcmaprpc.\n";
      }
      if (!input_dem.empty() && actual_session_type == "pinhole") {
        // User says PINHOLE but also gives a DEM.
        actual_session_type = "pinholemappinhole";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: pinholemappinhole.\n";
      }
      if (!input_dem.empty() && actual_session_type == "opticalbar") {
        // User says OPTICAL BAR but also gives a DEM.
        actual_session_type = "opticalbarmapopticalbar";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: opticalbarmapopticalbar.\n";
      }
      if (!input_dem.empty() && actual_session_type == "csm") {
        // User says CSM but also gives a DEM.
        // Mapprojection can happen either with csm or RPC cameras (the latter for DG)
        std::string cam_tag = "CAMERA_MODEL_TYPE";
        std::string l_cam_type = vw::cartography::read_header_string(left_image_file, cam_tag);
        if (l_cam_type == "rpc")
          actual_session_type = "csmmaprpc";
        else
          actual_session_type = "csmmapcsm"; // used also when l_cam_type is empty
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: " 
          << actual_session_type << ".\n";
      }
      if (!input_dem.empty() && actual_session_type == "isis") {
        // User says ISIS but also gives a DEM.
        actual_session_type = "isismapisis";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: isismapisis.\n";
      }
      if (!input_dem.empty() && actual_session_type == "spot5") {
        // User says SPOT5 but also gives a DEM.
        actual_session_type = "spot5maprpc";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: spot5maprpc.\n";
      }
      if (!input_dem.empty() && actual_session_type == "aster") {
        // User says ASTER but also gives a DEM.
        actual_session_type = "astermaprpc";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: astermaprpc.\n";
      }
      if (!input_dem.empty() && actual_session_type == "pleiades") {
        // User says Pleiades but also gives a DEM.
        actual_session_type = "pleiadesmappleiades";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: pleiadesmappleiades.\n";
      }
      
      // Quetly switch from nadirpinhole to pinhole for mapprojected images
      if (!input_dem.empty() && actual_session_type == "nadirpinhole") {
        // User says nadirpinhole but also gives a DEM.
        actual_session_type = "pinholemappinhole";
        VW_OUT(vw::DebugMessage,"asp") << "Changing session type to: pinhole.\n";
      }
      
    } // End map promotion section

    if (!input_dem.empty() &&
        (actual_session_type == "perusat")) {
      // User says PeruSat-1 or Pleiades but also gives a DEM, so the images were mapprojected.
      // If the mapprojection was done with the exact model, stereo becomes
      // painfully slow. If it was done with the RPC model, things become hard
      // to manage, and stereo needs to know both the exact and RPC model
      // and those are in different files. Hence, just don't allow mapprojected
      // images in this case.
      vw_throw(vw::NoImplErr() << "Stereo with mapprojected images and the PeruSat-1 or Pleiades "
               << "linescan model is not implemented. Use instead the RPC model.");
    }
    
    // We should know the session type by now.
    VW_ASSERT(!actual_session_type.empty(),
              vw::ArgumentErr() << "Could not determine stereo session type. "
              << "Please set it explicitly using the -t switch.\n"
              << "Options include: [nadirpinhole pinhole isis dg rpc spot5 aster perusat pleiades opticalbar csm pinholemappinhole isismapisis dgmaprpc rpcmaprpc spot5maprpc astermaprpc opticalbarmapopticalbar csmmapcsm csmmaprpc pleiadesmappleiades].\n");
    vw::vw_out() << "Using session: " << actual_session_type << "\n";

    // Compare the current session name to all recognized types
    // - Only one of these will ever get triggered
    StereoSession* session = NULL;
    if (actual_session_type == "dg")
      session = StereoSessionDG::construct();
    else if (actual_session_type == "dgmaprpc")
        session = StereoSessionDGMapRPC::construct();
    else if (actual_session_type == "nadirpinhole")
      session = StereoSessionNadirPinhole::construct();
    else if (actual_session_type == "pinhole")
      session = StereoSessionPinhole::construct();
    else if (actual_session_type == "rpc")
      session = StereoSessionRPC::construct();
    else if (actual_session_type == "rpcmaprpc")
      session = StereoSessionRPCMapRPC::construct();
    else if (actual_session_type == "pinholemappinhole")
      session = StereoSessionPinholeMapPinhole::construct();
    else if (actual_session_type == "opticalbarmapopticalbar")
      session = StereoSessionBarMapBar::construct();
    else if (actual_session_type == "spot5maprpc")
        session = StereoSessionSpot5MapRPC::construct();
    else if (actual_session_type == "astermaprpc")
        session = StereoSessionASTERMapRPC::construct();
    else if (actual_session_type == "pleiadesmappleiades")
        session = StereoSessionPleiadesMapPleiades::construct();
#if defined(ASP_HAVE_PKG_ISISIO) && ASP_HAVE_PKG_ISISIO == 1
    else if (actual_session_type == "isis")
      session = StereoSessionIsis::construct();
    else if (actual_session_type == "isismapisis")
      session = StereoSessionIsisMapIsis::construct();
#endif
    else if (actual_session_type == "spot5")
      session = StereoSessionSpot::construct();
    else if (actual_session_type == "perusat")
      session = StereoSessionPeruSat::construct();
    else if (actual_session_type == "pleiades")
      session = StereoSessionPleiades::construct();
    else if (actual_session_type == "aster")
      session = StereoSessionASTER::construct();
    else if (actual_session_type == "opticalbar")
      session = StereoSessionOpticalBar::construct();
    else if (actual_session_type == "csm")
      session = StereoSessionCsm::construct();
    else if (actual_session_type == "csmmapcsm")
      session = StereoSessionCsmMapCsm::construct();
    else if (actual_session_type == "csmmaprpc")
      session = StereoSessionCsmMapRpc::construct();
    if (session == 0)
      vw_throw(vw::NoImplErr() << "Unsupported stereo session type: " << actual_session_type);

    session->initialize(options,         // Initialize the new object
                        left_image_file,  right_image_file,
                        left_camera_file, right_camera_file,
                        out_prefix, input_dem);
    session_type = session->name(); // update the session name 
    return session;
} // End function create()

  
} // end namespace asp
