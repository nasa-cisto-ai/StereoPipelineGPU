// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// Camera utilities that need the stereo session

/// \file CameraUtils.cc

#include <asp/Sessions/CameraUtils.h>
#include <asp/Sessions/StereoSessionFactory.h>
#include <asp/Camera/CsmModel.h>

#include <vw/Core/Exception.h>
#include <vw/Camera/PinholeModel.h>
#include <vw/Camera/CameraUtilities.h>
#include <vw/InterestPoint/InterestData.h>

#include <string>
#include <iostream>

typedef boost::shared_ptr<asp::StereoSession> SessionPtr;

using namespace vw;

namespace asp {

// Load cameras from given image and camera files
void load_cameras(std::vector<std::string> const& image_files,
                  std::vector<std::string> const& camera_files,
                  std::string const& out_prefix, 
                  vw::GdalWriteOptions const& opt,
                  bool approximate_pinhole_intrinsics,
                  // Outputs
                  std::string & stereo_session, // may change
                  bool & single_threaded_cameras,
                  std::vector<boost::shared_ptr<vw::camera::CameraModel>> & camera_models) {

  // Initialize the outputs
  camera_models.clear();
  single_threaded_cameras = false;
  
  if (image_files.size() != camera_files.size()) 
    vw_throw(ArgumentErr() << "Expecting as many images as cameras.\n");  
  
  for (size_t i = 0; i < image_files.size(); i++) {
    vw_out(DebugMessage,"asp") << "Loading: " << image_files [i] << ' '
                               << camera_files[i] << "\n";
    
    // The same camera is double-loaded into the same session instance.
    // TODO: One day replace this with a simpler camera model loader class.
    // But note that this call also refines the stereo session name.
    SessionPtr session
      (asp::StereoSessionFactory::create(stereo_session, opt,
                                         image_files [i], image_files [i],
                                         camera_files[i], camera_files[i],
                                         out_prefix));
    
    camera_models.push_back(session->camera_model(image_files [i],
                                                  camera_files[i]));
    
    // This is necessary to avoid a crash with ISIS cameras which is single-threaded
    if (!session->supports_multi_threading())
      single_threaded_cameras = true;
    
    if (approximate_pinhole_intrinsics) {
      boost::shared_ptr<vw::camera::PinholeModel> pinhole_ptr = 
        boost::dynamic_pointer_cast<vw::camera::PinholeModel>(camera_models.back());
      // Replace lens distortion with fast approximation
      vw::camera::update_pinhole_for_fast_point2pixel<vw::camera::TsaiLensDistortion>
        (*(pinhole_ptr.get()), file_image_size(image_files[i]));
    }
  } // End loop through images loading all the camera models
  
  return;
}

// Find the datum based on cameras. For stereo session pinhole will return WGS84.
void datum_from_cameras(std::vector<std::string> const& image_files,
                        std::vector<std::string> const& camera_files,
                        std::string & stereo_session, // may change
                        // Outputs
                        vw::cartography::Datum & datum) {
  
  datum.set_well_known_datum("WGS84"); // if no luck
  std::string out_prefix = "run";

  // Look for a non-pinole camera, as a pinhole camera does not have a datum
  for (size_t i = 0; i < image_files.size(); i++) {

    // This is for the case when there is a mix of pinhole and non-pinhole
    // cameras. In that case, the pinhole cameras will be ignored. Must
    // reset the session to be able to load the non-pinhole cameras.
    if (stereo_session == "pinhole")
      stereo_session = "";

    SessionPtr session(asp::StereoSessionFactory::create(stereo_session, // may change
                                                         vw::GdalWriteOptions(),
                                                         image_files [i], image_files [i],
                                                         camera_files[i], camera_files[i],
                                                         out_prefix)); 
    if (stereo_session == "pinhole")
      continue;

    bool use_sphere_for_non_earth = true;
    datum = session->get_datum(session->camera_model(image_files [i],
                                                     camera_files[i]).get(),
                               use_sphere_for_non_earth);
    break; // found the datum
  }

  return;
}
  
} // end namespace asp
