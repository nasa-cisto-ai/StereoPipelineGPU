#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file StereoSession.cc
///

#include <asp/Sessions/StereoSession.h>
#include <asp/Core/BundleAdjustUtils.h>
#include <asp/Camera/AdjustedLinescanDGModel.h>
#include <asp/Camera/RPCModel.h>
#include <asp/Core/AspStringUtils.h>

#include <vw/Core/Exception.h>
#include <vw/Core/Log.h>
#include <vw/Math/Vector.h>
#include <vw/Math/Functors.h>
#include <vw/Math/Geometry.h>
#include <vw/Image/PixelMask.h>
#include <vw/Image/PixelTypeInfo.h>
#include <vw/FileIO/DiskImageResource.h>
#include <vw/FileIO/DiskImageView.h>
#include <vw/Cartography/GeoReferenceUtils.h>
#include <vw/Cartography/Map2CamTrans.h>
#include <vw/FileIO/MatrixIO.h>
#include <vw/Core/Stopwatch.h>
#include <boost/filesystem/operations.hpp>

#include <map>
#include <utility>
#include <string>
#include <ostream>
#include <limits>

using namespace vw;
using namespace vw::cartography;

namespace asp {

  // Pass over all the string variables we use
  void StereoSession::initialize(vw::GdalWriteOptions const& options,
                                 std::string const& left_image_file,
                                 std::string const& right_image_file,
                                 std::string const& left_camera_file,
                                 std::string const& right_camera_file,
                                 std::string const& out_prefix,
                                 std::string const& input_dem) {
    m_options           = options;
    m_left_image_file   = left_image_file;
    m_right_image_file  = right_image_file;
    m_left_camera_file  = left_camera_file;
    m_right_camera_file = right_camera_file;
    m_out_prefix        = out_prefix;
    m_input_dem         = input_dem;
    
    // Do any other initialization steps needed
    init_disk_transform();
  }

  // Init the transform that is used to undo the mapprojection. The header
  // file of the mapprojected images contain a lot of info that we load
  // along the way.
  void StereoSession::init_disk_transform() {

    if (!isMapProjected()) // Nothing to do for non map-projected types.
      return;

    // Back up the bundle-adjust prefix that should be used only with the
    // original camera model, not with the model used in mapprojection
    // (e.g., the original camera model could have been DG, but in
    // map-projection we could have used RPC).
    std::string ba_pref_bk = stereo_settings().bundle_adjust_prefix;

    // Load the name of the camera model, session, and DEM used in mapprojection
    // based on the record in that image. Load the bundle adjust prefix from the
    // mapprojected image. It can be empty, when such a prefix was not used in
    // mapprojection. 
    std::string l_adj_prefix, r_adj_prefix, l_image_file, r_image_file,
    l_cam_type, r_cam_type, l_cam_file, r_cam_file, l_dem_file, r_dem_file;

    {
      std::string adj_key = "BUNDLE_ADJUST_PREFIX", 
      img_file_key = "INPUT_IMAGE_FILE", cam_type_key = "CAMERA_MODEL_TYPE",
        cam_file_key = "CAMERA_FILE", dem_file_key = "DEM_FILE"; 
      // We only read our own map projections which are written in GDAL format
      boost::shared_ptr<vw::DiskImageResource>
        l_rsrc(new vw::DiskImageResourceGDAL(m_left_image_file));
      boost::shared_ptr<vw::DiskImageResource>
        r_rsrc(new vw::DiskImageResourceGDAL(m_right_image_file));
      vw::cartography::read_header_string(*l_rsrc.get(), adj_key,      l_adj_prefix);
      vw::cartography::read_header_string(*l_rsrc.get(), img_file_key, l_image_file);
      vw::cartography::read_header_string(*l_rsrc.get(), cam_type_key, l_cam_type);
      vw::cartography::read_header_string(*l_rsrc.get(), cam_file_key, l_cam_file);
      vw::cartography::read_header_string(*l_rsrc.get(), dem_file_key, l_dem_file);

      vw::cartography::read_header_string(*r_rsrc.get(), adj_key,      r_adj_prefix);
      vw::cartography::read_header_string(*r_rsrc.get(), img_file_key, r_image_file);
      vw::cartography::read_header_string(*r_rsrc.get(), cam_type_key, r_cam_type);
      vw::cartography::read_header_string(*r_rsrc.get(), cam_file_key, r_cam_file);
      vw::cartography::read_header_string(*r_rsrc.get(), dem_file_key, r_dem_file);
    }

    // Sanity checks, throw an error
    if (l_adj_prefix != r_adj_prefix)
      vw_throw(ArgumentErr()  << "The left and right mapprojected image bundle adjust "
        << "prefixes do not match. Got: \" " << l_adj_prefix << "\" and \"" 
        << r_adj_prefix << "\".\n");
    if (l_cam_type != r_cam_type)
      vw_throw(ArgumentErr() << "The left and right mapprojected image camera types "
        << "do not match. Got: \" " << l_cam_type << "\" and \"" 
        << r_cam_type << "\".\n");

    // If l_cam_type is empty, and session name is XmapY, use cam type Y.
    if (l_cam_type == "") {
      std::string tri_cam_type, mapproj_cam_type; 
      asp::parseCamTypes(this->name(), tri_cam_type, mapproj_cam_type);
      vw::vw_out(WarningMessage)
           << "The camera type was not specified in the mapprojected "
           << "images header. Assuming mapprojection was done with camera type: "
           << mapproj_cam_type << ".\n";
      l_cam_type = mapproj_cam_type;
      r_cam_type = mapproj_cam_type;
    }

    // We will use the camera file from the mapprojected image to undo the
    // mapprojection, not the one specified by the user, which is used in
    // triangulation. We check for l_image_file and r_image_file,
    // because the camera files can be empty, like for .cub and rpc.
    std::string curr_left_camera_file = m_left_camera_file;
    std::string curr_right_camera_file = m_right_camera_file;
    if (l_image_file != "" && r_image_file != "") {
      curr_left_camera_file  = l_cam_file;
      curr_right_camera_file = r_cam_file;

      // The DEM the user provided better be the one used for map projection.
      if (m_input_dem != l_dem_file || m_input_dem != r_dem_file)
        vw_throw(ArgumentErr() << "The DEM used for map projection is different "
          << "from the one provided on the command line.\n"
          << "Left image DEM:   " << l_dem_file << "\n"
          << "Right image DEM:  " << r_dem_file << "\n"
          << "Command line DEM: " << m_input_dem << "\n");
    }

    // When loading camera models from the image files, we either use the sensor model for
    // the current session type or else the RPC model which is often used as an approximation.
    const Vector2 zero_pixel_offset(0,0);
    stereo_settings().bundle_adjust_prefix = "";
    if (l_adj_prefix != "" && l_adj_prefix != "NONE")
      stereo_settings().bundle_adjust_prefix = l_adj_prefix;

    if (l_cam_type == "rpc") {
      // This message is useful, because sometimes there is confusion as to whether
      // the RPC model or original model is used in mapprojection.
      vw_out() << "Loading RPC cameras used in mapprojection.\n";
      m_left_map_proj_model = load_rpc_camera_model(m_left_image_file,  curr_left_camera_file,
                                                    zero_pixel_offset);
    } else { // Use the native model
      vw_out() << "Loading " << l_cam_type << " cameras used in mapprojection.\n";
      m_left_map_proj_model = load_camera_model(m_left_image_file,  curr_left_camera_file,
                                                zero_pixel_offset);
    }
    vw_out() << "Mapprojected images bundle adjustment prefix: \"" 
              << stereo_settings().bundle_adjust_prefix << "\"\n";
    
    stereo_settings().bundle_adjust_prefix = "";
    if (r_adj_prefix != "" && r_adj_prefix != "NONE")
      stereo_settings().bundle_adjust_prefix = r_adj_prefix;
    if (r_cam_type == "rpc")
      m_right_map_proj_model = load_rpc_camera_model(m_right_image_file, curr_right_camera_file,
                                                     zero_pixel_offset);
    else // Use the native model
      m_right_map_proj_model = load_camera_model(m_right_image_file, curr_right_camera_file,
                                                 zero_pixel_offset);

    // These are useful messages
    vw_out() << "Left camera file used in mapprojection: " << curr_left_camera_file << "\n";
    vw_out() << "Right camera file used in mapprojection: " << curr_right_camera_file << "\n";

    // Go back to the original bundle-adjust prefix now that we have
    // loaded the models used in map-projection.
    stereo_settings().bundle_adjust_prefix = ba_pref_bk;

    VW_ASSERT( m_left_map_proj_model.get() && m_right_map_proj_model.get(),
              ArgumentErr() << "StereoSession: Unable to locate map "
              << "projection camera model inside input files!" );

    // Double check that we can read the DEM and that it has cartographic information.
    VW_ASSERT(!m_input_dem.empty(), InputErr() << "StereoSession: Require input DEM." );
    if (!boost::filesystem::exists(m_input_dem))
      vw_throw(ArgumentErr() << "StereoSession: DEM '" << m_input_dem << "' does not exist.");
  }

  // Peek inside the images and camera models and return the datum and projection,
  // or at least the datum, packaged in a georef.
  vw::cartography::GeoReference StereoSession::get_georef() {

    vw::cartography::GeoReference georef;
    
    // First try to see if the image is map-projected.
    bool has_georef = read_georeference(georef, m_left_image_file);

    bool has_datum = false;
    vw::cartography::Datum datum;
    if (!stereo_settings().correlator_mode) {
      has_datum = true;
      boost::shared_ptr<vw::camera::CameraModel> cam = this->camera_model(m_left_image_file,
                                                                          m_left_camera_file);
      // Spherical datum for non-Earth, as done usually. Used
      // consistently this way in bundle adjustment and stereo.
      bool use_sphere_for_non_earth = true; 
      datum = this->get_datum(cam.get(), use_sphere_for_non_earth);
    }
    
    // TODO(oalexan1): If the datum read from the image and the one read from the
    // session disagree, what to do? 
    
    if (!has_georef) {
      // The best we can do is to get the datum, even non-projected
      // images have that. Create however a fake valid georeference to
      // go with this datum, otherwise we can't read the datum when we
      // needed it later.

      georef = vw::cartography::GeoReference();
      Matrix3x3 transform = georef.transform();

      // assume these are degrees, does not mater much, but it needs be small enough
      double small = 1e-8;
      transform(0,0) = small;
      transform(1,1) = small;
      transform(0,2) = small;
      transform(1,2) = small;
      georef.set_transform(transform);

      georef.set_geographic();
      
      if (has_datum)
        georef.set_datum(datum);
    }

    return georef;
  }

  // Default implementation of this function.  Derived classes will probably override this.
  void StereoSession::camera_models(boost::shared_ptr<vw::camera::CameraModel> &cam1,
                                    boost::shared_ptr<vw::camera::CameraModel> &cam2) {
    cam1 = camera_model(m_left_image_file,  m_left_camera_file);
    cam2 = camera_model(m_right_image_file, m_right_camera_file);
  }

boost::shared_ptr<vw::camera::CameraModel>
StereoSession::camera_model(std::string const& image_file, std::string const& camera_file,
                            bool quiet) {

  if (stereo_settings().correlator_mode) {
    // No cameras exist, so make some dummy cameras. Recall that we
    // set the session to rpc in this mode so that it is assumed that
    // the cameras may hiding in the images rather than kept
    // separately. In this mode the cameras should not actually get
    // used.
    vw::Vector<double, 20> v; v[0] = 1.0; // to a void a zero denominator
    vw::Vector2 v2(1.0, 1.0); // to avoid division by 0
    vw::Vector3 v3(1.0, 1.0, 1.0);
    boost::shared_ptr<vw::camera::CameraModel>
      cam(new RPCModel(vw::cartography::Datum("WGS84"), v, v, v, v, v2, v2, v3, v3));
    return cam;
  }
  
  // If the desired camera is already loaded, do not load it again.
  std::pair<std::string, std::string> image_cam_pair = std::make_pair(image_file, camera_file);
  
  auto map_it = m_camera_model.find(image_cam_pair);
  if (map_it != m_camera_model.end()) 
    return map_it->second;

  // Sometime when we do many attempts at loading cameras we don't want to print
  // this message. 
  if (!quiet) 
    vw_out() << "Loading camera model: " << image_file << ' ' << camera_file << "\n";

  // Retrieve the pixel offset (if any) to cropped images
  vw::Vector2 pixel_offset = camera_pixel_offset(m_input_dem,
                                                 m_left_image_file,
                                                 m_right_image_file,
                                                 image_file);
  
  if (camera_file == "") // No camera file provided, use the image file.
    m_camera_model[image_cam_pair] = load_camera_model(image_file, image_file, pixel_offset);
  else // Camera file provided
    m_camera_model[image_cam_pair] = load_camera_model(image_file, camera_file, pixel_offset);

  return m_camera_model[image_cam_pair];
}

// Default preprocessing hook. Some sessions may override it.
void StereoSession::preprocessing_hook(bool adjust_left_image_size,
                                       std::string const& left_input_file,
                                       std::string const& right_input_file,
                                       std::string      & left_output_file,
                                       std::string      & right_output_file) {

  std::string left_cropped_file, right_cropped_file;
  vw::GdalWriteOptions options;
  float left_nodata_value, right_nodata_value;
  bool has_left_georef, has_right_georef;
  vw::cartography::GeoReference left_georef, right_georef;
  bool exit_early =
    StereoSession::shared_preprocessing_hook(options,
                                             left_input_file,   right_input_file,
                                             left_output_file,  right_output_file,
                                             left_cropped_file, right_cropped_file,
                                             left_nodata_value, right_nodata_value,
                                             has_left_georef,   has_right_georef,
                                             left_georef,       right_georef);

  if (exit_early)
    return;
  
  // Load the images (can be cropped or original ones)
  DiskImageView<float> left_disk_image(left_cropped_file);
  DiskImageView<float> right_disk_image(right_cropped_file);
  
  // Get the image sizes. Later alignment options can choose to
  // change this parameters (such as affine epipolar alignment).
  Vector2i left_size  = file_image_size(left_cropped_file);
  Vector2i right_size = file_image_size(right_cropped_file);
  
  // Set up image masks
  ImageViewRef<PixelMask<float>> left_masked_image
    = create_mask_less_or_equal(left_disk_image,  left_nodata_value);
  ImageViewRef<PixelMask<float>> right_masked_image
    = create_mask_less_or_equal(right_disk_image, right_nodata_value);
  
  // Compute input image statistics. This can be slow so use a timer.
  vw::Stopwatch sw1;
  sw1.start();
  Vector6f left_stats  = gather_stats(left_masked_image,  "left",
                                      this->m_out_prefix, left_cropped_file);
  sw1.stop();  
  vw_out() << "Left image stats time: " << sw1.elapsed_seconds() << std::endl;
  vw::Stopwatch sw2;
  sw2.start();  
  Vector6f right_stats = gather_stats(right_masked_image, "right",
                                      this->m_out_prefix, right_cropped_file);
  sw2.stop();
  vw_out() << "Right image stats time: " << sw2.elapsed_seconds() << std::endl;
  ImageViewRef<PixelMask<float>> Limg, Rimg;
  
  // Use no-data in interpolation and edge extension
  PixelMask<float>nodata_pix(0); nodata_pix.invalidate();
  ValueEdgeExtension<PixelMask<float>> ext_nodata(nodata_pix); 
  
  // Initialize alignment matrices and get the input image sizes.
  Matrix<double> align_left_matrix  = math::identity_matrix<3>(),
    align_right_matrix = math::identity_matrix<3>();
  
  // Generate aligned versions of the input images according to the
  // options.
  vw_out() << "\t--> Applying alignment method: " << stereo_settings().alignment_method << "\n";
  if (stereo_settings().alignment_method == "epipolar") {
    
    epipolar_alignment(left_masked_image, right_masked_image, ext_nodata,  
                       // Outputs
                       Limg, Rimg);
    
  } else if (stereo_settings().alignment_method == "homography"     ||
             stereo_settings().alignment_method == "affineepipolar" ||
             stereo_settings().alignment_method == "local_epipolar") {
    
    // Load the cameras
    boost::shared_ptr<camera::CameraModel> left_cam, right_cam;
    this->camera_models(left_cam, right_cam);
    
    determine_image_alignment(// Inputs
                              m_out_prefix, left_cropped_file, right_cropped_file,  
                              left_input_file,
                              left_stats, right_stats, left_nodata_value, right_nodata_value,  
                              left_cam, right_cam,
                              adjust_left_image_size,  
                              // In-out
                              align_left_matrix, align_right_matrix, left_size, right_size);
    
    // Apply the alignment transform to both input images
    Limg = transform(left_masked_image,
                     HomographyTransform(align_left_matrix),
                     left_size.x(), left_size.y());
    Rimg = transform(right_masked_image,
                     HomographyTransform(align_right_matrix),
                     right_size.x(), right_size.y());
      
  } else {
    // No alignment, just provide the original files.
    Limg = left_masked_image;
    Rimg = right_masked_image;
  } // End of image alignment block
  
  // Apply our normalization options.
  bool use_percentile_stretch = false;
  bool do_not_exceed_min_max = (this->name() == "isis" ||
                                this->name() == "isismapisis");
  // TODO(oalexan1): Should one add above "csm" and "csmmapcsm" / "csmmaprpc"?
  asp::normalize_images(stereo_settings().force_use_entire_range,
                        stereo_settings().individually_normalize,
                        use_percentile_stretch, 
                        do_not_exceed_min_max,
                        left_stats, right_stats, Limg, Rimg);

  if (stereo_settings().alignment_method == "local_epipolar") {
    // Save these stats for local epipolar alignment, as they will be used
    // later in each tile.
    std::string left_stats_file  = this->m_out_prefix + "-lStats.tif";
    std::string right_stats_file = this->m_out_prefix + "-rStats.tif";
    vw_out() << "Writing: " << left_stats_file << ' ' << right_stats_file << std::endl;
    vw::Vector<float32> left_stats2  = left_stats;  // cast
    vw::Vector<float32> right_stats2 = right_stats; // cast
    write_vector(left_stats_file,  left_stats2 );
    write_vector(right_stats_file, right_stats2);
  }
  
  // The output no-data value must be < 0 as we scale the images to [0, 1].
  bool has_nodata = true;
  float output_nodata = -32768.0;
  vw_out() << "\t--> Writing pre-aligned images.\n";
  vw_out() << "\t--> Writing: " << left_output_file << ".\n";
  vw::Stopwatch sw3;
  sw3.start();
  block_write_gdal_image(left_output_file, apply_mask(Limg, output_nodata),
                         has_left_georef, left_georef,
                         has_nodata, output_nodata, options,
                         TerminalProgressCallback("asp","\t  L:  "));
  sw3.stop();
  vw_out() << "Time to write left image: " << sw3.elapsed_seconds() << std::endl;
    
  vw_out() << "\t--> Writing: " << right_output_file << ".\n";
  vw::Stopwatch sw4;
  sw4.start();
  if (stereo_settings().alignment_method == "none") {
    // Do not crop the right image to have the same dimensions as the
    // left image. Since there is no alignment, and images may not be
    // georeferenced, we do not know what portion of the right image
    // corresponds best to the left image, so cropping may throw away
    // an area where the left and right images overlap.
    block_write_gdal_image(right_output_file, apply_mask(Rimg, output_nodata),
                           has_right_georef, right_georef,
                           has_nodata, output_nodata, options,
                           TerminalProgressCallback("asp","\t  R:  "));
  } else {
    // Crop the right aligned image consistently with how the alignment
    // transform expects it. The resulting R.tif will have the same
    // size as L.tif. Extra pixels will be filled with nodata.
    block_write_gdal_image(right_output_file,
                           apply_mask(crop(edge_extend(Rimg, ext_nodata), 
                                           bounding_box(Limg)), output_nodata),
                           has_right_georef, right_georef,
                           has_nodata, output_nodata, options,
                           TerminalProgressCallback("asp","\t  R:  ") );
  }
  sw4.stop();
  vw_out() << "Time to write right image: " << sw4.elapsed_seconds() << std::endl;

  // For bathy runs only
  if (this->do_bathymetry())
    this->align_bathy_masks(options);

} // End function preprocessing_hook
  
void StereoSession::pre_filtering_hook(std::string const& input_file,
                                       std::string      & output_file) {
  output_file = input_file;
}

ImageViewRef<PixelMask<Vector2f> >
StereoSession::pre_pointcloud_hook(std::string const& input_file) {
  return DiskImageView<PixelMask<Vector2f> >( input_file );
}

// A little function whose goal is to avoid repeating same logic in a handful of places
void crop_bathy_mask(vw::GdalWriteOptions const& options,
                     std::string const& input_mask_file, std::string const& input_image_file,
                     BBox2i const& crop_win, std::string const& cropped_mask_file) {
  
  if (input_mask_file == "") 
    vw_throw( ArgumentErr() << "Required bathy mask file was not specified.");

  // Sanity check, input image and mask must have same size
  DiskImageView<float> input_image(input_image_file);
  DiskImageView<float> input_bathy_mask(input_mask_file);
  if (input_bathy_mask.cols() != input_image.cols() ||
      input_bathy_mask.rows() != input_image.rows()) 
    vw_throw(ArgumentErr() << "Input image and input bathy mask don't have the same dimensions.");
  
  float mask_nodata_value = -std::numeric_limits<float>::max();
  if (!vw::read_nodata_val(input_mask_file, mask_nodata_value))
    vw_throw(ArgumentErr() << "Unable to read the nodata value from " << input_mask_file);

  vw::cartography::GeoReference georef;
  bool has_georef = read_georeference(georef, input_image_file);

  bool has_mask_nodata = true;
  vw_out() << "\t--> Writing cropped mask: " << cropped_mask_file << "\n";
  block_write_gdal_image(cropped_mask_file,
                         crop(input_bathy_mask, crop_win),
                         has_georef, crop(georef, crop_win),
                         has_mask_nodata, mask_nodata_value,
                         options,
                         TerminalProgressCallback("asp", "\t:  "));
}
  
bool StereoSession::
shared_preprocessing_hook(vw::GdalWriteOptions & options,
                          std::string const                 & left_input_file,
                          std::string const                 & right_input_file,
                          std::string                       & left_output_file,
                          std::string                       & right_output_file,
                          std::string                       & left_cropped_file,
                          std::string                       & right_cropped_file,
                          float                             & left_nodata_value,
                          float                             & right_nodata_value,
                          bool                              & has_left_georef,
                          bool                              & has_right_georef,
                          vw::cartography::GeoReference     & left_georef,
                          vw::cartography::GeoReference     & right_georef){

  // Retrieve nodata values and let the handles go out of scope right away.
  // For this to work the ISIS type must be registered with the
  // DiskImageResource class. This happens in "stereo.cc", so
  // these calls will create DiskImageResourceIsis objects.
  {
    boost::shared_ptr<DiskImageResource>
      left_rsrc (DiskImageResourcePtr(left_input_file )),
      right_rsrc(DiskImageResourcePtr(right_input_file));
    asp::get_nodata_values(left_rsrc,         right_rsrc,
                           left_nodata_value, right_nodata_value);
  }

  // Set output file paths
  left_output_file  = this->m_out_prefix + "-L.tif";
  right_output_file = this->m_out_prefix + "-R.tif";

  // Enforce no predictor in compression, it works badly with L.tif and R.tif.
  options = this->m_options;
  options.gdal_options["PREDICTOR"] = "1";

  // Read the georef if available in the input images
  has_left_georef  = read_georeference(left_georef,  left_input_file);
  has_right_georef = read_georeference(right_georef, right_input_file);
  if ( stereo_settings().alignment_method != "none") {
    // If any alignment at all happens, the georef will be messed up.
    has_left_georef = false;
    has_right_georef = false;
  }

  bool crop_left  = (stereo_settings().left_image_crop_win  != BBox2i(0, 0, 0, 0));
  bool crop_right = (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0));

  // Here either the input image or the cropped images will be returned,
  // depending on whether the crop actually happens
  left_cropped_file = this->left_cropped_image();
  right_cropped_file = this->right_cropped_image();
  
  // If the output files already exist and are newer than the input files,
  // and we don't crop both left and right images, then there is nothing to do here.
  // Note: Must make sure all outputs are initialized before we
  // get to this part where we exit early.
  
  bool do_bathy = StereoSession::do_bathymetry();

  std::vector<std::string> check_files;
  check_files.push_back(left_input_file);
  check_files.push_back(right_input_file);
  check_files.push_back(m_left_camera_file);
  check_files.push_back(m_right_camera_file);
  bool rebuild = (!is_latest_timestamp(left_output_file, check_files) ||
                  !is_latest_timestamp(right_output_file, check_files));

  if (do_bathy) {
    rebuild = (rebuild ||
               (!is_latest_timestamp(left_aligned_bathy_mask(), check_files) ||
                !is_latest_timestamp(right_aligned_bathy_mask(), check_files)));
  }
  
  if (!rebuild && !crop_left && !crop_right) {
    try {
      vw_log().console_log().rule_set().add_rule(-1, "fileio");
      DiskImageView<PixelGray<float32> > out_left (left_output_file );
      DiskImageView<PixelGray<float32> > out_right(right_output_file);

      if (do_bathy) {
        DiskImageView<float> left_bathy_mask (left_aligned_bathy_mask());
        DiskImageView<float> right_bathy_mask(right_aligned_bathy_mask());
      }
      
      vw_out(InfoMessage) << "\t--> Using cached normalized input images.\n";
      vw_settings().reload_config();
      return true; // Return true if we exist early since the images exist
    } catch (vw::ArgumentErr const& e) {
      // This throws on a corrupted file.
      vw_settings().reload_config();
    } catch (vw::IOErr const& e) {
      vw_settings().reload_config();
    }
  } // End check for existing output files
  
  // See if to crop the images
  if (crop_left) {
    // Crop and save the left image to left_cropped_file
    has_left_georef = read_georeference(left_georef, left_input_file);
    bool has_nodata = true;

    DiskImageView<float> left_orig_image(left_input_file);
    BBox2i left_win = stereo_settings().left_image_crop_win;
    left_win.crop(bounding_box(left_orig_image));

    ImageViewRef<float> left_cropped_image = crop(left_orig_image, left_win);

    if (stereo_settings().left_image_clip != "") {
      // Replace the crop with a given clip. This is a very rarely used option.
      // It can be handy when investigating CCD artifacts correction.
      left_cropped_image = DiskImageView<float>(stereo_settings().left_image_clip);
      if (left_cropped_image.cols() != left_win.width() ||
          left_cropped_image.rows() != left_win.height() ) {
        vw_throw( ArgumentErr() << "The image specified via --left-image-clip has different "
                  << "dimensions than set via --left-image-crop-win.");
      }
    }
    
    vw_out() << "\t--> Writing cropped image: " << left_cropped_file << "\n";
    block_write_gdal_image(left_cropped_file,
                           left_cropped_image,
                           has_left_georef, crop(left_georef, left_win),
                           has_nodata, left_nodata_value,
                           options,
                           TerminalProgressCallback("asp", "\t:  "));
  }
  
  if (crop_right) {
    // Crop the right image and write to right_cropped_file
    has_right_georef = read_georeference(right_georef, right_input_file);
    bool has_nodata = true;

    DiskImageView<float> right_orig_image(right_input_file);
    BBox2i right_win = stereo_settings().right_image_crop_win;
    right_win.crop(bounding_box(right_orig_image));

    ImageViewRef<float> right_cropped_image = crop(right_orig_image, right_win);

    if (stereo_settings().right_image_clip != "") {
      // Replace the crop with a given clip. This is a very rarely used option.
      // It can be handy when investigating CCD artifacts correction.
      right_cropped_image = DiskImageView<float>(stereo_settings().right_image_clip);
      if (right_cropped_image.cols() != right_win.width() ||
          right_cropped_image.rows() != right_win.height() ) {
        vw_throw( ArgumentErr() << "The image specified via --right-image-clip has different "
                  << "dimensions than set via --right-image-crop-win.");
      }
    }
    
    vw_out() << "\t--> Writing cropped image: " << right_cropped_file << "\n";
    block_write_gdal_image(right_cropped_file,
                           right_cropped_image,
                           has_right_georef,
                           crop(right_georef, right_win),
                           has_nodata, right_nodata_value,
                           options,
                           TerminalProgressCallback("asp", "\t:  "));
  }
  
  // Re-read the georef, since it may have changed above.
  has_left_georef  = read_georeference(left_georef,  left_cropped_file);
  has_right_georef = read_georeference(right_georef, right_cropped_file);
  if ( stereo_settings().alignment_method != "none") {
    // If any alignment at all happens, the georef will be messed up.
    has_left_georef  = false;
    has_right_georef = false;
  }
  
  return false; // don't exit early
}

void StereoSession::read_bathy_masks(float & left_bathy_nodata,  float & right_bathy_nodata, 
                                     vw::ImageViewRef<vw::PixelMask<float>> & left_bathy_mask,
                                     vw::ImageViewRef<vw::PixelMask<float>> & right_bathy_mask) {
  
  std::string left_cropped_mask_file = left_cropped_bathy_mask();
  left_bathy_nodata = -std::numeric_limits<float>::max();
  if (!vw::read_nodata_val(left_cropped_mask_file, left_bathy_nodata))
    vw_throw(ArgumentErr() << "Unable to read the nodata value from "
             << left_cropped_mask_file);
  left_bathy_mask = create_mask(DiskImageView<float>(left_cropped_mask_file),
                                left_bathy_nodata);

  std::string right_cropped_mask_file = right_cropped_bathy_mask();
  right_bathy_nodata = -std::numeric_limits<float>::max();
  if (!vw::read_nodata_val(right_cropped_mask_file, right_bathy_nodata))
    vw_throw(ArgumentErr() << "Unable to read the nodata value from "
             << right_cropped_mask_file);
  right_bathy_mask = create_mask(DiskImageView<float>(right_cropped_mask_file),
                                right_bathy_nodata);
  
  // The left image (after crop) better needs to have the same dims
  // as the left mask after crop, and same for the right
  DiskImageView<float> left_image(this->left_cropped_image());
  DiskImageView<float> right_image(this->right_cropped_image());
  if (left_bathy_mask.cols() != left_image.cols()   || 
      left_bathy_mask.rows() != left_image.rows()   || 
      right_bathy_mask.cols() != right_image.cols() || 
      right_bathy_mask.rows() != right_image.rows() ) {
    vw_throw( ArgumentErr() << "The dimensions of bathymetry masks don't agree "
              << "with the image sizes (after crop win, if applicable)." );
  }
  
}

void StereoSession::read_aligned_bathy_masks
(vw::ImageViewRef<vw::PixelMask<float>> & left_aligned_bathy_mask_image,
 vw::ImageViewRef<vw::PixelMask<float>> & right_aligned_bathy_mask_image) {

  std::string left_aligned_mask_file = left_aligned_bathy_mask();
  float left_bathy_nodata = -std::numeric_limits<float>::max();
  if (!vw::read_nodata_val(left_aligned_mask_file, left_bathy_nodata))
    vw_throw(ArgumentErr() << "Unable to read the nodata value from "
             << left_aligned_mask_file);
  left_aligned_bathy_mask_image = create_mask(DiskImageView<float>(left_aligned_mask_file),
                                              left_bathy_nodata);
  
  std::string right_aligned_mask_file = right_aligned_bathy_mask();
  float right_bathy_nodata = -std::numeric_limits<float>::max();
  if (!vw::read_nodata_val(right_aligned_mask_file, right_bathy_nodata))
    vw_throw(ArgumentErr() << "Unable to read the nodata value from "
             << right_aligned_mask_file);
  right_aligned_bathy_mask_image = create_mask(DiskImageView<float>(right_aligned_mask_file),
                                right_bathy_nodata);
}

bool StereoSession::do_bathymetry() const {
  return (stereo_settings().left_bathy_mask != "" || 
          stereo_settings().right_bathy_mask != "");
}

// Align the bathy masks. This will be called in stereo_pprc and, if
// needed, in stereo_tri. Skip this if the masks already exit and are
// not older than the images. This code mirrors very closely the logic
// for how the images are aligned.
void StereoSession::align_bathy_masks(vw::GdalWriteOptions const& options) {

  bool do_bathy = StereoSession::do_bathymetry();
  
  if (!do_bathy)
    return;

  // Check the timestamp of aligned masks
  std::vector<std::string> check_files;
  check_files.push_back(m_left_image_file);
  check_files.push_back(m_right_image_file);
  check_files.push_back(m_left_camera_file);
  check_files.push_back(m_right_camera_file);
  check_files.push_back(stereo_settings().left_bathy_mask);
  check_files.push_back(stereo_settings().right_bathy_mask);
  
  bool crop_left  = (stereo_settings().left_image_crop_win  != BBox2i(0, 0, 0, 0));
  bool crop_right = (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0));
  
  bool rebuild = (!is_latest_timestamp(left_aligned_bathy_mask(), check_files) ||
                  !is_latest_timestamp(right_aligned_bathy_mask(), check_files));
  
  if (!rebuild && !crop_left && !crop_right) {
    try {
      vw_log().console_log().rule_set().add_rule(-1, "fileio");
      DiskImageView<float> left_bathy_mask (left_aligned_bathy_mask());
      DiskImageView<float> right_bathy_mask(right_aligned_bathy_mask());
      
      vw_out(InfoMessage) << "\t--> Using cached aligned bathy masks.\n";
      vw_settings().reload_config();
      return; // no need to rebuild, the results exit and are good
      
    } catch (vw::ArgumentErr const& e) {
      // This throws on a corrupted file.
      vw_settings().reload_config();
    } catch (vw::IOErr const& e) {
      vw_settings().reload_config();
    }
  } // End check for existing output files

  // See if to crop the masks
  if (crop_left) {
    DiskImageView<float> left_orig_image(m_left_image_file);
    BBox2i left_win = stereo_settings().left_image_crop_win;
    left_win.crop(bounding_box(left_orig_image));
    crop_bathy_mask(options, stereo_settings().left_bathy_mask,  
                    m_left_image_file, left_win, left_cropped_bathy_mask());
  }
  if (crop_right) {
    DiskImageView<float> right_orig_image(m_right_image_file);
    BBox2i right_win = stereo_settings().right_image_crop_win;
    right_win.crop(bounding_box(right_orig_image));
    crop_bathy_mask(options, stereo_settings().right_bathy_mask,  
                    m_right_image_file, right_win, right_cropped_bathy_mask());
  }
  
  // Read the unaligned cropped masks
  ImageViewRef<PixelMask<float>> left_bathy_mask, right_bathy_mask;
  float left_bathy_nodata = -std::numeric_limits<float>::max();
  float right_bathy_nodata = -std::numeric_limits<float>::max();
  StereoSession::read_bathy_masks(left_bathy_nodata, right_bathy_nodata,
                                  left_bathy_mask, right_bathy_mask);

  // Use no-data in interpolation and edge extension.
  PixelMask<float>bathy_nodata_pix(0); bathy_nodata_pix.invalidate();
  ValueEdgeExtension<PixelMask<float>> bathy_ext_nodata(bathy_nodata_pix); 

  // Get the aligned size from the images already aligned
  Vector2i left_size;
  std::string left_aligned_file = this->m_out_prefix + "-L.tif";
  if (boost::filesystem::exists(left_aligned_file))
    left_size = file_image_size(left_aligned_file);
  else
    vw_throw(NoImplErr() << "Could not read: " << left_aligned_file);

  // Read alignment matrices
  Matrix<double> align_left_matrix = math::identity_matrix<3>();
  std::string left_matrix_file = this->m_out_prefix + "-align-L.exr";
  if (stereo_settings().alignment_method == "affineepipolar" ||
      stereo_settings().alignment_method == "local_epipolar") {
    if (boost::filesystem::exists(left_matrix_file))
      read_matrix(align_left_matrix, left_matrix_file);
    else
      vw_throw(NoImplErr() << "Could not read: " << left_matrix_file);
  }

  Matrix<double> align_right_matrix = math::identity_matrix<3>();
  std::string right_matrix_file = this->m_out_prefix + "-align-R.exr";
  if (stereo_settings().alignment_method == "homography"     ||
      stereo_settings().alignment_method == "affineepipolar" ||
      stereo_settings().alignment_method == "local_epipolar") {
    if (boost::filesystem::exists(right_matrix_file))
      read_matrix(align_right_matrix, right_matrix_file);
    else
      vw_throw(NoImplErr() << "Could not read " << right_matrix_file);
  }

  // Generate aligned versions of the masks according to the options.
  ImageViewRef<PixelMask<float>> left_aligned_bathy_mask, right_aligned_bathy_mask;
  if (stereo_settings().alignment_method == "homography"     ||
      stereo_settings().alignment_method == "affineepipolar" ||
      stereo_settings().alignment_method == "local_epipolar") {
    
    left_aligned_bathy_mask = transform(left_bathy_mask,
                                        HomographyTransform(align_left_matrix),
                                        left_size.x(), left_size.y());

    // Note how we use left_size and not right_size
    right_aligned_bathy_mask = transform(right_bathy_mask,
                                         HomographyTransform(align_right_matrix),
                                         left_size.x(), left_size.y());
    
  } else if (stereo_settings().alignment_method == "none") {
    // No alignment
    left_aligned_bathy_mask  = left_bathy_mask;
    right_aligned_bathy_mask = right_bathy_mask;
  } // End of image alignment block

  bool has_bathy_nodata = true;
  float output_nodata = -32768.0;
  
  // Read the georef of the cropped left and right images saved before the masks
  vw::cartography::GeoReference left_georef, right_georef;
  bool has_left_georef  = read_georeference(left_georef,  this->left_cropped_image());
  bool has_right_georef = read_georeference(right_georef, this->right_cropped_image());
  
  std::string left_aligned_bathy_mask_file = StereoSession::left_aligned_bathy_mask();
  vw_out() << "\t--> Writing: " << left_aligned_bathy_mask_file << ".\n";
  block_write_gdal_image(left_aligned_bathy_mask_file,
                         apply_mask(left_aligned_bathy_mask, left_bathy_nodata),
                         has_left_georef, left_georef,
                         has_bathy_nodata, left_bathy_nodata, options,
                         TerminalProgressCallback("asp","\t  L bathy mask:  "));

  // Use same logic as when the right aligned image is written
  if (stereo_settings().alignment_method == "none") {
    std::string right_aligned_bathy_mask_file = StereoSession::right_aligned_bathy_mask();
    vw_out() << "\t--> Writing: " << right_aligned_bathy_mask_file << ".\n";
    block_write_gdal_image(right_aligned_bathy_mask_file,
                           apply_mask(right_aligned_bathy_mask, right_bathy_nodata),
                           has_right_georef, right_georef,
                           has_bathy_nodata, right_bathy_nodata, options,
                           TerminalProgressCallback("asp","\t  R bathy mask:  "));
  } else {
    std::string right_aligned_bathy_mask_file = StereoSession::right_aligned_bathy_mask();
    vw_out() << "\t--> Writing: " << right_aligned_bathy_mask_file << ".\n";
    block_write_gdal_image(right_aligned_bathy_mask_file,
                           apply_mask(crop(edge_extend(right_aligned_bathy_mask,
                                                       bathy_ext_nodata),
                                           // Note how we use the left aligned mask bbox
                                           bounding_box(left_aligned_bathy_mask)),
                                      right_bathy_nodata),
                           has_right_georef, right_georef,
                           has_bathy_nodata, right_bathy_nodata,
                           options,
                           TerminalProgressCallback("asp","\t  R bathy mask:  ") );
  }
}
  
// Return the left and right cropped images. These are the same
// as the input images unless the cropping is on.
std::string StereoSession::left_cropped_image() const{
  std::string cropped_image = m_left_image_file;
  if (stereo_settings().left_image_crop_win != BBox2i(0, 0, 0, 0))
    cropped_image = m_out_prefix + "-L-cropped.tif";
  return cropped_image;
}
  
std::string StereoSession::right_cropped_image() const{
  std::string cropped_image = m_right_image_file;
  if (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0))
    cropped_image = m_out_prefix + "-R-cropped.tif";
  return cropped_image;
}
  
// Apply epipolar alignment to images, if the camera models are pinhole. This will
// be reimplemented in StereoSessionPinhole.
void StereoSession::epipolar_alignment(vw::ImageViewRef<vw::PixelMask<float>> left_masked_image,
                                       vw::ImageViewRef<vw::PixelMask<float>> right_masked_image,
                                       vw::ValueEdgeExtension<vw::PixelMask<float>> ext_nodata,
                                       // Outputs
                                       vw::ImageViewRef<vw::PixelMask<float>> & Limg, 
                                       vw::ImageViewRef<vw::PixelMask<float>> & Rimg) {
  vw_throw(ArgumentErr() << "Epipolar alignment is only implemented for pinhole cameras.");
}
  
std::string StereoSession::left_cropped_bathy_mask() const {
  if (!do_bathymetry()) 
    vw_throw( ArgumentErr() << "The left cropped bathy mask is requested when "
              << "bathymetry mode is not on." );

  bool crop_left = (stereo_settings().left_image_crop_win != BBox2i(0, 0, 0, 0));
  if (!crop_left) 
    return stereo_settings().left_bathy_mask;

  return this->m_out_prefix + "-L_cropped_bathy_mask.tif";
}
  
std::string StereoSession::right_cropped_bathy_mask() const {
  if (!do_bathymetry()) 
    vw_throw( ArgumentErr() << "The right cropped bathy mask is requested when "
              << "bathymetry mode is not on." );

  bool crop_right = (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0));
  if (!crop_right) 
    return stereo_settings().right_bathy_mask;

  return this->m_out_prefix + "-R_cropped_bathy_mask.tif";
}

std::string StereoSession::left_aligned_bathy_mask() const {
  return m_out_prefix + "-L_aligned_bathy_mask.tif";
}
  
std::string StereoSession::right_aligned_bathy_mask() const {
  return m_out_prefix + "-R_aligned_bathy_mask.tif";
}

void StereoSession::get_input_image_crops(vw::BBox2i &left_image_crop,
                                          vw::BBox2i &right_image_crop) const {

  // Set the ROIs to the entire image if the input crop windows are not set.
  Vector2i left_size  = file_image_size(m_left_image_file );
  Vector2i right_size = file_image_size(m_right_image_file);

  if (stereo_settings().left_image_crop_win != BBox2i(0, 0, 0, 0))
    left_image_crop  = stereo_settings().left_image_crop_win;
  else
    left_image_crop = BBox2i(0, 0, left_size [0], left_size [1]);

  if (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0))
    right_image_crop = stereo_settings().right_image_crop_win;
  else
    right_image_crop = BBox2i(0, 0, right_size[0], right_size[1]);
}


//------------------------------------------------------------------------------
// Code for handling disk-to-sensor transform


// TODO: Move this function somewhere else!
/// Computes a Map2CamTrans given a DEM, image, and a sensor model.
inline StereoSession::tx_type
getTransformFromMapProject(const std::string &input_dem_path,
                           const std::string &img_file_path,
                           boost::shared_ptr<vw::camera::CameraModel> map_proj_model_ptr) {

  // Read in data necessary for the Map2CamTrans object
  cartography::GeoReference dem_georef, image_georef;
  if (!read_georeference(dem_georef, input_dem_path))
    vw_throw( ArgumentErr() << "The DEM \"" << input_dem_path
              << "\" lacks georeferencing information.");
  if (!read_georeference(image_georef, img_file_path))
    vw_throw( ArgumentErr() << "The image \"" << img_file_path
              << "\" lacks georeferencing information.");

  bool call_from_mapproject = false;
  DiskImageView<float> img(img_file_path);
  return StereoSession::tx_type(new cartography::Map2CamTrans(map_proj_model_ptr.get(),
                                   image_georef, dem_georef, input_dem_path,
                                   Vector2(img.cols(), img.rows()),
                                   call_from_mapproject));
}

typename StereoSession::tx_type
StereoSession::tx_left_homography() const {
  Matrix<double> tx = math::identity_matrix<3>();
  if ( stereo_settings().alignment_method == "homography" ||
       stereo_settings().alignment_method == "affineepipolar" ||
       stereo_settings().alignment_method == "local_epipolar" ) {
    read_matrix( tx, m_out_prefix + "-align-L.exr" );
  }
  return tx_type( new vw::HomographyTransform(tx) );
}

typename StereoSession::tx_type
StereoSession::tx_right_homography() const {
  Matrix<double> tx = math::identity_matrix<3>();
  if ( stereo_settings().alignment_method == "homography" ||
       stereo_settings().alignment_method == "affineepipolar" ||
       stereo_settings().alignment_method == "local_epipolar" ) {
    read_matrix( tx, m_out_prefix + "-align-R.exr" );
  }
  return tx_type( new vw::HomographyTransform(tx) );
}

typename StereoSession::tx_type
StereoSession::tx_identity() const {
  Matrix<double> tx = math::identity_matrix<3>();
  return tx_type( new vw::HomographyTransform(tx) );
}


typename StereoSession::tx_type
StereoSession::tx_left_map_trans() const {
  std::string left_map_proj_image = this->left_cropped_image();
  if (!m_left_map_proj_model)
    vw_throw( ArgumentErr() << "Map projection model not loaded for image "
              << left_map_proj_image);
  return getTransformFromMapProject(m_input_dem, left_map_proj_image, m_left_map_proj_model);
}
typename StereoSession::tx_type
StereoSession::tx_right_map_trans() const {
  std::string right_map_proj_image = this->right_cropped_image();
  if (!m_right_map_proj_model)
    vw_throw( ArgumentErr() << "Map projection model not loaded for image "
              << right_map_proj_image);
  return getTransformFromMapProject(m_input_dem, right_map_proj_image, m_right_map_proj_model);
}


// Load an RPC model. Any adjustment in stereo_settings().bundle_adjust_prefix
// will be applied.
boost::shared_ptr<vw::camera::CameraModel> 
StereoSession::load_rpc_camera_model(std::string const& image_file, 
                                     std::string const& camera_file,
                                     Vector2 pixel_offset) const {

  std::string err1, err2;
  try {
    if (camera_file != ""){
      return load_adjusted_model(m_camera_loader.load_rpc_camera_model(camera_file),
                                image_file, camera_file, pixel_offset);
    }
  }
  catch(std::exception const& e1) {
    err1 = e1.what();
  }
  try {
    return load_adjusted_model(m_camera_loader.load_rpc_camera_model(image_file),
                              image_file, camera_file, pixel_offset);
  }
  catch(std::exception const& e2) {
    err2 = e2.what();
  }

  // For Cartosat, GDAL chokes. The user must move {image}_RPC_ORG.TXT
  // to {image}_RPC.TXT
  std::string truncated =  boost::filesystem::path(image_file).replace_extension("").string();
  std::string org_file = truncated + "_RPC_ORG.TXT";
  std::string rpc_file = truncated + "_RPC.TXT";
  std::string msg = "";
  if (boost::filesystem::exists(org_file))
    msg = "Detected file: " + org_file + ". If this is Cartosat data, any such files "
      "must be moved to names like " + rpc_file
      + " by overwriting those files if necessary.\n";
    
  // Raise a custom exception if both failed
  vw_throw(ArgumentErr() << "Unable to load RPC model from either " << image_file
          << " or " << camera_file << ".\n"
	   << err1 << "\n" << err2 << "\n" << msg);
} // End function load_rpc_camera_model


vw::Vector2 StereoSession::camera_pixel_offset(std::string const& input_dem,
                                               std::string const& left_image_file,
                                               std::string const& right_image_file,
                                               std::string const& curr_image_file){
  // For map-projected images we don't apply a pixel offset.
  // When we need to do stereo on cropped images, we just
  // crop the images together with their georeferences.
  if (input_dem != "")
    return Vector2();

  bool crop_left  = (stereo_settings().left_image_crop_win  != BBox2i(0, 0, 0, 0));
  bool crop_right = (stereo_settings().right_image_crop_win != BBox2i(0, 0, 0, 0));
  vw::Vector2 left_pixel_offset, right_pixel_offset;
  if (crop_left ) left_pixel_offset  = stereo_settings().left_image_crop_win.min();
  if (crop_right) right_pixel_offset = stereo_settings().right_image_crop_win.min();
  
  if (curr_image_file == left_image_file)
    return left_pixel_offset;
  else if (curr_image_file == right_image_file)
    return right_pixel_offset;
  else
    // If the image files were not specified, no offset and no error.
    if ((left_image_file != "") || (right_image_file != ""))
      vw_throw(ArgumentErr() << "Supplied image file does not match left or right image file.");

  return Vector2();
}

boost::shared_ptr<vw::camera::CameraModel>
StereoSession::load_adjusted_model(boost::shared_ptr<vw::camera::CameraModel> cam,
                                  std::string const& image_file,
                                  std::string const& camera_file,
                                  vw::Vector2 const& pixel_offset){

  // Any tool using adjusted camera models must pre-populate the
  // prefix at which to find them.
  std::string ba_pref = stereo_settings().bundle_adjust_prefix;
  if (ba_pref == "" && pixel_offset == vw::Vector2())
    return cam; // Return the unadjusted cameras if there is no adjustment

  std::vector<Vector3> position_correction;
  std::vector<Quat   > pose_correction;

  // These must start initialized. Note that we may have a pixel
  // offset passed in from outside, or a pixel offset and scale
  // that we read from an adjust file. We will throw an error
  // below if both scenarios happen.
  Vector2 local_pixel_offset = pixel_offset;
  double local_scale = 1.0;

  // Ensure these vectors are populated even when there are no corrections to read,
  // as we may still have pixel offset.
  position_correction.push_back(Vector3());
  pose_correction.push_back(Quat(math::identity_matrix<3>()));

  if (ba_pref != "") { // If a bundle adjustment file was specified

    // Get full BA file path
    std::string adjust_file = asp::bundle_adjust_file_name(ba_pref, image_file, camera_file);

    if (!boost::filesystem::exists(adjust_file))
      vw_throw(InputErr() << "Missing adjusted camera model: " << adjust_file << ".\n");

    vw_out() << "Using adjusted camera model: " << adjust_file << std::endl;
    bool piecewise_adjustments;
    Vector2 adjustment_bounds;
    std::string session;
    asp::read_adjustments(adjust_file, piecewise_adjustments,
                          adjustment_bounds, position_correction, pose_correction,
			  local_pixel_offset, local_scale, // these will change
			  session);

    if (local_pixel_offset != Vector2() || local_scale != 1.0) {
      // We read a custom scale and pixel offset passed by the user. But then
      // the pixel offset passed in by the caller is not valid. Instead of
      // sorting things out simply refuse to allow this scenario.
      if (pixel_offset != Vector2()) {
        vw_throw(InputErr() << "Cannot use crop win functionality with custom "
                 << "scale and pixel offset in .adjust files.\n");
      }
    }else{
      // In this case we have local_pixel_offset == (0, 0) local_scale == 1.0.
      // So use the pixel_offset passed in by the caller. Scale will stay at 1.0.
      local_pixel_offset = pixel_offset;
    }
    
    if (position_correction.empty() || pose_correction.empty())
      vw_throw(InputErr() << "Unable to read corrections.\n");

    // Handle the case of piecewise adjustments for DG and other cameras
    if (piecewise_adjustments) {

      DiskImageView<float> img(image_file);
      Vector2i image_size(img.cols(), img.rows());

      if ( session == "dg" || session == "dgmaprpc") {

        // Create the adjusted DG model
        boost::shared_ptr<camera::CameraModel> adj_dg_cam
          (new AdjustedLinescanDGModel(cam,
                                       stereo_settings().piecewise_adjustment_interp_type,
                                       adjustment_bounds, position_correction,
                                       pose_correction, image_size));

        // Apply the pixel offset and pose corrections. So this a second adjustment
        // on top of the first.
        boost::shared_ptr<camera::CameraModel> adj_dg_cam2
          (new vw::camera::AdjustedCameraModel(adj_dg_cam, Vector3(),
                                               Quat(math::identity_matrix<3>()), pixel_offset));

        return adj_dg_cam2;
      }else{
         // Create the generic adjusted model
         boost::shared_ptr<camera::CameraModel> adj_generic_cam
           (new PiecewiseAdjustedLinescanModel(cam,
                                               stereo_settings().piecewise_adjustment_interp_type,
                                               adjustment_bounds, position_correction,
                                               pose_correction, image_size));

         // Apply the pixel offset and pose corrections. So this a second adjustment
         // on top of the first.
         boost::shared_ptr<camera::CameraModel> adj_generic_cam2
           (new vw::camera::AdjustedCameraModel(adj_generic_cam, Vector3(),
                                                Quat(math::identity_matrix<3>()), pixel_offset));

         return adj_generic_cam2;
      }

    } // End case for piecewise DG adjustment

  } // End case for parsing bundle adjustment file

  // Create VW adjusted camera model object with the info we loaded
  return boost::shared_ptr<camera::CameraModel>(new vw::camera::AdjustedCameraModel
                                                (cam, position_correction[0],
                                                 pose_correction[0], local_pixel_offset,
						 local_scale));
}

/// Function to apply a functor to each pixel of an input image.
/// Traverse the image row by row.
template <class ViewT, class FuncT>
void for_each_pixel_rowwise(const vw::ImageViewBase<ViewT> &view_, FuncT &func, 
  vw::TerminalProgressCallback const& progress) {
  using namespace vw;
  const ViewT& view = view_.impl();
  typedef typename ViewT::pixel_accessor pixel_accessor;
  pixel_accessor plane_acc = view.origin();

  for (int32 plane = view.planes(); plane; plane--) { // Loop through planes

    pixel_accessor row_acc = plane_acc;
    for (int32 row = 0; row<view.rows(); row++) { // Loop through rows
      progress.report_fractional_progress(row, view.rows());
      pixel_accessor col_acc = row_acc;
      for (int32 col = view.cols(); col; col--) { // Loop along the row
        func(*col_acc);  // Apply the functor to this pixel value
        col_acc.next_col();
      }
      row_acc.next_row();
    }
    plane_acc.next_plane();
  }
  progress.report_finished();
}

/// Function to apply a functor to each pixel of an input image.
/// Traverse the image column by column.
template <class ViewT, class FuncT>
void for_each_pixel_columnwise(const vw::ImageViewBase<ViewT> &view_, FuncT &func, 
  vw::TerminalProgressCallback const& progress) {
  using namespace vw;
  const ViewT& view = view_.impl();
  typedef typename ViewT::pixel_accessor pixel_accessor;
  pixel_accessor plane_acc = view.origin();

  for (int32 plane = view.planes(); plane; plane--) { // Loop through planes
    
    pixel_accessor col_acc = plane_acc;
    for (int32 col = 0; col < view.cols(); col++) { // Loop through cols
      progress.report_fractional_progress(col, view.cols());
      pixel_accessor row_acc = col_acc;
      for (int32 row = view.rows(); row; row--) { // Loop along cols
        func(*row_acc);  // Apply the functor to this pixel value
        row_acc.next_row();
      }
      col_acc.next_col();
    }
    plane_acc.next_plane();
  }
  progress.report_finished();

  return;
}

// Compute the min, max, mean, and standard deviation of an image object and
// write them to a log. This is not a member function.
// - "tag" is only used to make the log messages more descriptive.
// - If prefix and image_path is set, will cache the results to a file.
// For efficiency, the image must be traversed either rowwise or columnwise,
// depending on how it is stored on disk.
vw::Vector6f gather_stats(vw::ImageViewRef<vw::PixelMask<float>> image, 
                          std::string const& tag,
                          std::string const& prefix, 
                          std::string const& image_path) {

  using namespace vw;
  namespace fs = boost::filesystem;
  Vector6f result;

  vw_out(InfoMessage) << "Computing statistics for " + tag << std::endl;

  const bool use_cache = ((prefix != "") && (image_path != ""));
  std::string cache_path = "";
  if (use_cache) {
    if (image_path.find(prefix) == 0) {
      // If the image is, for example, run/run-L.tif,
      // then cache_path = run/run-L-stats.tif.
      cache_path =  fs::change_extension(image_path, "").string() + "-stats.tif";
    }else {
      // If the image is left_image.tif, 
      // then cache_path = run/run-left_image.tif
      cache_path = prefix + '-' + fs::path(image_path).stem().string() + "-stats.tif";
    }
  }
  
  // Check if this stats file was computed after any image modifications.
  if ((use_cache && asp::is_latest_timestamp(cache_path, image_path)) ||
      (stereo_settings().force_reuse_match_files && fs::exists(cache_path))) {
    vw_out(InfoMessage) << "\t--> Reading statistics from file " + cache_path << std::endl;
    Vector<float32> stats;
    read_vector(stats, cache_path); // Just fetch the stats from the file on disk.
    result = stats;

  } else { // Compute the results

    // Read the resource and determine the block structure on disk. Use a boost shared ptr.
    vw::Vector2i block_size;
    {
      boost::shared_ptr<DiskImageResource> rsrc (DiskImageResourcePtr(image_path));
      block_size  = rsrc->block_read_size();
    }
    // print a warning that procesing can be slow if any of the block size coords are bigger than 5120
    if (block_size[0] > 5120 || block_size[1] > 5120) {
      vw_out(WarningMessage) << "Image " << image_path 
        << " has block sizes of dimensions " << block_size[0] << " x " << block_size[1] 
        << " (as shown by gdalinfo). This can make processing slow. Consider converting "
        << "it to tile format, using the command:\n" 
        << "gdal_translate -co TILED=yes -co BLOCKXSIZE=256 -co BLOCKYSIZE=256 " 
        << "input.tif output.tif\n";
    }

    // Compute statistics at a reduced resolution
    const float TARGET_NUM_PIXELS = 1000000;
    float num_pixels = float(image.cols())*float(image.rows());
    int   stat_scale = int(ceil(sqrt(num_pixels / TARGET_NUM_PIXELS)));

    vw_out(InfoMessage) << "Using downsample scale: " << stat_scale << std::endl;

    ChannelAccumulator<vw::math::CDFAccumulator<float> > accumulator;
    vw::TerminalProgressCallback tp("asp","\t  stats:  ");
    if (block_size[0] >= block_size[1]) // Rows are long, so go row by row
     for_each_pixel_rowwise(subsample( edge_extend(image, ConstantEdgeExtension()),
                               stat_scale), accumulator, tp);
    else // Columns are long, so go column by column
     for_each_pixel_columnwise(subsample( edge_extend(image, ConstantEdgeExtension()),
                                  stat_scale), accumulator, tp);

    result[0] = accumulator.quantile(0); // Min
    result[1] = accumulator.quantile(1); // Max
    result[2] = accumulator.approximate_mean();
    result[3] = accumulator.approximate_stddev();
    result[4] = accumulator.quantile(0.02); // Percentile values
    result[5] = accumulator.quantile(0.98);

    // Cache the results to disk
    if (use_cache) {
      vw_out() << "\t    Writing stats file: " << cache_path << std::endl;
      Vector<float32> stats = result;  // cast
      write_vector(cache_path, stats);
    }

  } // Done computing the results

  vw_out(InfoMessage) << "\t    " << tag << ": [ lo: " << result[0] << " hi: " << result[1]
                      << " mean: " << result[2] << " std_dev: "  << result[3] << " ]\n";

  return result;
}

} // End namespace asp
