#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

/// \file StereoSessionGdal.cc
///
#include <vw/Image/ImageMath.h>
#include <vw/Image/Manipulation.h>
#include <vw/Image/MaskViews.h>
#include <vw/Image/Transform.h>
#include <vw/Camera/Extrinsics.h>
#include <vw/Math/Matrix.h>
#include <vw/Cartography/Datum.h>

#include <asp/Core/StereoSettings.h>
#include <asp/Core/AffineEpipolar.h>
#include <asp/Sessions/StereoSessionGdal.h>
#include <asp/Sessions/StereoSessionASTER.h>
#include <asp/Sessions/StereoSessionNadirPinhole.h>
#include <asp/Camera/RPCModel.h>
#include <asp/Camera/RPC_XML.h>

#include <boost/shared_ptr.hpp>
#include <boost/filesystem/operations.hpp>
namespace fs = boost::filesystem;

#include <iostream>
#include <string>
#include <vector>
#include <algorithm>

using namespace vw;

namespace asp {

/// Returns the target datum to use for a given camera model
vw::cartography::Datum StereoSessionCsm::get_datum(const vw::camera::CameraModel* cam,
                                                   bool use_sphere_for_non_earth) const {
  
  // Peek at the .cub file to get the planet name without reading
  // it as an ISIS camera (which can fail unless the ISISDATA
  // folder exists, and for CSM that is not guaranteed.)
  // The CSM camera .json file itself lacks this information.
  std::string spheroid_name = asp::read_target_name(m_left_image_file);
  std::string datum_name = "D_" + spheroid_name; // may be refined later

  const asp::CsmModel * cast_csm_cam
    = dynamic_cast<const asp::CsmModel*>(vw::camera::unadjusted_model(cam));
  VW_ASSERT(cast_csm_cam != NULL,
            vw::ArgumentErr() << "Could not load a CSM camera.\n");

  // Read the ellipsoid radii
  vw::Vector3 radii = cast_csm_cam->target_radii();
  double radius1 = (radii[0] + radii[1]) / 2; // average the x and y axes (semi-major) 
  double radius2 = radius1;

  // Auto-guess the datum if not available
  vw::cartography::Datum wgs84("WGS84");
  vw::cartography::Datum moon("D_MOON");
  vw::cartography::Datum mars("D_MARS");
  bool is_wgs84 = (std::abs(wgs84.semi_major_axis() - radius1)  < 1e-7 &&
                   std::abs(wgs84.semi_minor_axis() - radii[2]) < 1e-7);
  bool is_moon =  (std::abs(moon.semi_major_axis()  - radius1)  < 1e-7 &&
                   std::abs(moon.semi_minor_axis()  - radii[2]) < 1e-7);
  bool is_mars =  (std::abs(mars.semi_major_axis()  - radius1)  < 1e-7 &&
                   std::abs(mars.semi_minor_axis()  - radii[2]) < 1e-7);
  
  if (boost::to_lower_copy(spheroid_name).find("unknown") != std::string::npos) {
    // Unknown datum. Try to fill in the name from above.
    if (is_wgs84)
      return wgs84;
    if (is_moon)
      return moon;
    if (is_mars)
      return mars;
  }
  
  // For Earth always use two radii. The logic below should distinguish Venus.
  bool has_earth_radius = (std::abs(radius1/wgs84.semi_major_axis() - 1.0) < 0.05);
  if (!use_sphere_for_non_earth || has_earth_radius)
    radius2 = radii[2]; // let the semi-minor axis be distinct from the semi-major axis
  
  vw::cartography::Datum datum(datum_name, spheroid_name,
                               "Reference Meridian", radius1, radius2, 0);
  
  return datum;
}
  
} // End namespace asp

