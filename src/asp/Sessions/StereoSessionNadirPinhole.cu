// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file StereoSessionNadirPinhole.cc
///
#include <vw/Image/ImageMath.h>
#include <vw/Image/Manipulation.h>
#include <vw/Image/MaskViews.h>
#include <vw/Image/Transform.h>
#include <vw/Camera/Extrinsics.h>
#include <vw/Math/Matrix.h>
#include <vw/Cartography/Datum.h>

#include <asp/Core/StereoSettings.h>
#include <asp/Core/AffineEpipolar.h>
#include <asp/Sessions/StereoSessionGdal.h>
#include <asp/Sessions/StereoSessionASTER.h>
#include <asp/Sessions/StereoSessionNadirPinhole.h>
#include <asp/Camera/RPCModel.h>
#include <asp/Camera/RPC_XML.h>

#include <boost/shared_ptr.hpp>
#include <boost/filesystem/operations.hpp>
namespace fs = boost::filesystem;

#include <iostream>
#include <string>
#include <vector>
#include <algorithm>

using namespace vw;
using namespace asp;

namespace asp {

} // End namespace asp
