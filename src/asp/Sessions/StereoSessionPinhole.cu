// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file StereoSessionPinhole.cc
///

#include <asp/Sessions/StereoSessionPinhole.h>
#include <vw/Math/BBox.h>
#include <vw/Math/Geometry.h>
#include <vw/Math/Matrix.h>
#include <vw/Math/RANSAC.h>
#include <vw/Math/Vector.h>
#include <vw/Image/ImageViewRef.h>
#include <vw/Image/MaskViews.h>
#include <vw/Camera/CameraUtilities.h>
#include <vw/Stereo/DisparityMap.h>

#include <boost/shared_ptr.hpp>
#include <boost/filesystem/operations.hpp>
namespace fs = boost::filesystem;

using namespace vw;
using namespace vw::camera;

// This class assumes pinhole cameras with no datum, such as on a rover.
// For pinhole satellite images the nadirpinhole mode is suggested.

namespace asp {

boost::shared_ptr<vw::camera::CameraModel>
StereoSessionPinhole::load_camera_model
  (std::string const& image_file, std::string const& camera_file, Vector2 pixel_offset) const{

  return load_adj_pinhole_model(image_file, camera_file,
                                m_left_image_file,  m_right_image_file,
                                m_left_camera_file, m_right_camera_file,
                                m_input_dem);
}

// Apply epipolar alignment to images, if the camera models are pinhole
void StereoSessionPinhole::
epipolar_alignment(vw::ImageViewRef<vw::PixelMask<float>> left_masked_image,
                   vw::ImageViewRef<vw::PixelMask<float>> right_masked_image,
                   vw::ValueEdgeExtension<vw::PixelMask<float>> ext_nodata,
                   // Outputs
                   vw::ImageViewRef<vw::PixelMask<float>> & Limg, 
                   vw::ImageViewRef<vw::PixelMask<float>> & Rimg) {

  // Load the two images and fetch the two camera models
  boost::shared_ptr<camera::CameraModel> left_cam, right_cam;

  std::string lcase_file = boost::to_lower_copy(m_left_camera_file);
  if (boost::ends_with(lcase_file, ".pinhole") || boost::ends_with(lcase_file, ".tsai")) {
      
    // This loads epipolar-aligned camera models.
    // - The out sizes incorporate the crop amount if any, the camera models 
    Vector2i left_out_size, right_out_size;
    load_camera_models(left_cam, right_cam, left_out_size, right_out_size);
      
    // Get the input image crop regions, if any.
    BBox2i left_image_in_roi, right_image_in_roi;
    get_input_image_crops(left_image_in_roi, right_image_in_roi);

    // Write out the camera models used to generate the aligned images.
    // - Currently this won't work if we used .adjust files from bundle_adjust.
    PinholeModel* left_pin_model  = dynamic_cast<PinholeModel*>(left_cam.get ());
    PinholeModel* right_pin_model = dynamic_cast<PinholeModel*>(right_cam.get());
    if (left_pin_model)
      left_pin_model->write(m_out_prefix + "-L.tsai");
    if (right_pin_model)
      right_pin_model->write(m_out_prefix + "-R.tsai");

    // Transform the input images to be as if they were captured by the
    //  epipolar-aligned camera models, aligning the two images.
    get_epipolar_transformed_pinhole_images(m_left_camera_file, m_right_camera_file,
                                            left_cam, right_cam,
                                            left_masked_image, right_masked_image,
                                            left_image_in_roi, right_image_in_roi,
                                            left_out_size, right_out_size,
                                            Limg, Rimg,
                                            ext_nodata,
                                            BilinearInterpolation());

  } else { // Handle CAHV derived models
    camera_models(left_cam, right_cam);
    get_epipolar_transformed_images(m_left_camera_file, m_right_camera_file,
                                    left_cam, right_cam,
                                    left_masked_image, right_masked_image,
                                    Limg, Rimg, ext_nodata);
  }
}

void StereoSessionPinhole::get_unaligned_camera_models(
                                 boost::shared_ptr<vw::camera::CameraModel> &left_cam,
                                 boost::shared_ptr<vw::camera::CameraModel> &right_cam) const{

  // Retrieve the pixel offset (if any) to cropped images
  vw::Vector2 left_pixel_offset  = camera_pixel_offset(m_input_dem, m_left_image_file,
                                                       m_right_image_file, m_left_image_file);
  vw::Vector2 right_pixel_offset = camera_pixel_offset(m_input_dem, m_left_image_file,
                                                       m_right_image_file, m_right_image_file);

  // Load the camera models adjusted for cropping
  left_cam  = load_adjusted_model(vw::camera::load_pinhole_camera_model(m_left_camera_file),
                                  m_left_image_file, m_left_camera_file, left_pixel_offset);
  right_cam = load_adjusted_model(vw::camera::load_pinhole_camera_model(m_right_camera_file),
                                  m_right_image_file, m_right_camera_file, right_pixel_offset);
}


boost::shared_ptr<vw::camera::CameraModel>
StereoSessionPinhole::load_adj_pinhole_model(std::string const& image_file,
                                             std::string const& camera_file,
                                             std::string const& left_image_file,
                                             std::string const& right_image_file,
                                             std::string const& left_camera_file,
                                             std::string const& right_camera_file,
                                             std::string const& input_dem){

  // Unfortunately the pinhole case is more complicated since the left
  // and right files are inter-dependent.

  // Retrieve the pixel offset (if any) to cropped images
  vw::Vector2 pixel_offset = camera_pixel_offset(input_dem,
                                                 left_image_file,
                                                 right_image_file,
                                                 image_file);
  
  if ( stereo_settings().alignment_method != "epipolar" ) {
    // Not epipolar, just load the camera model.
    return load_adjusted_model(vw::camera::load_pinhole_camera_model(camera_file),
                               image_file, camera_file, pixel_offset);
  }
  // Otherwise handle the epipolar case

  bool is_left_camera = true;
  if (image_file == left_image_file)
    is_left_camera = true;
  else if (image_file == right_image_file)
    is_left_camera = false;
  else
    (ArgumentErr() << "StereoSessionPinhole: supplied camera model filename "
     << "does not match the name supplied in the constructor.");

  std::string lcase_file = boost::to_lower_copy(left_camera_file);
  if (boost::ends_with(lcase_file, ".pinhole") || boost::ends_with(lcase_file, ".tsai")) {
    // Use PinholeModel epipolar code

    PinholeModel left_pin (left_camera_file );
    PinholeModel right_pin(right_camera_file);

    // Create epipolar rectified camera views
    boost::shared_ptr<PinholeModel> epipolar_left_pin (new PinholeModel);
    boost::shared_ptr<PinholeModel> epipolar_right_pin(new PinholeModel);
    epipolar(left_pin,  right_pin, *epipolar_left_pin, *epipolar_right_pin);

    // Expand epipolar cameras to contain the entire source images.
    Vector2i left_size  = file_image_size(left_image_file );
    Vector2i right_size = file_image_size(right_image_file);
    Vector2i epi_size1, epi_size2; // TODO: Use these!
    resize_epipolar_cameras_to_fit(left_pin, right_pin,
                                   *(epipolar_left_pin.get()), *(epipolar_right_pin.get()),
                                   BBox2i(Vector2i(0,0), left_size),
                                   BBox2i(Vector2i(0,0), right_size),
                                   epi_size1, epi_size2);

    if (is_left_camera)
      return load_adjusted_model(epipolar_left_pin, image_file, camera_file, pixel_offset);
    // Right camera
    return load_adjusted_model(epipolar_right_pin, image_file, camera_file, pixel_offset);
       
  } else { // Not PinholeModel, use CAHV epipolar code.

    // Fetch CAHV version of the two input pinhole files
    boost::shared_ptr<CAHVModel> left_cahv
      = vw::camera::load_cahv_pinhole_camera_model(left_image_file,  left_camera_file );
    boost::shared_ptr<CAHVModel> right_cahv
      = vw::camera::load_cahv_pinhole_camera_model(right_image_file, right_camera_file);

    // Create epipolar rectified camera views
    boost::shared_ptr<CAHVModel> epipolar_left_cahv (new CAHVModel);
    boost::shared_ptr<CAHVModel> epipolar_right_cahv(new CAHVModel);
    epipolar(*(left_cahv.get()),  *(right_cahv.get()),
             *epipolar_left_cahv, *epipolar_right_cahv);

    if (is_left_camera)
      return load_adjusted_model(epipolar_left_cahv, image_file, camera_file, pixel_offset);
    // Right camera
    return load_adjusted_model(epipolar_right_cahv, image_file, camera_file, pixel_offset);
  }
}

void StereoSessionPinhole::camera_models(boost::shared_ptr<vw::camera::CameraModel> &cam1,
                                         boost::shared_ptr<vw::camera::CameraModel> &cam2) {
  vw::Vector2i left_out_size, right_out_size;
  load_camera_models(cam1, cam2, left_out_size, right_out_size);
}

void StereoSessionPinhole::load_camera_models(
                   boost::shared_ptr<vw::camera::CameraModel> &left_cam,
                   boost::shared_ptr<vw::camera::CameraModel> &right_cam,
                   Vector2i &left_out_size, Vector2i &right_out_size) {

  std::string lcase_file = boost::to_lower_copy(m_left_camera_file);
  if ( (stereo_settings().alignment_method != "epipolar") ||
       ( !boost::ends_with(lcase_file, ".pinhole") &&
         !boost::ends_with(lcase_file, ".tsai"   )   ) ) {
    // Non-PinholeModel and non-epipolar case, just use the simpler handling method
    // and leave the sizes unset, they won't be used.
    left_cam  = camera_model(m_left_image_file,  m_left_camera_file );
    right_cam = camera_model(m_right_image_file, m_right_camera_file);
    return;
  }

  // PinholeModel case is more complicated. The camera models
  // returned do not include a crop offset, but the aligned camera
  // models have been shifted so that they are aligned after the crop
  // has been applied.

  PinholeModel left_pin (m_left_camera_file );
  PinholeModel right_pin(m_right_camera_file);

  // Create epipolar rectified camera views
  boost::shared_ptr<PinholeModel> epipolar_left_pin (new PinholeModel);
  boost::shared_ptr<PinholeModel> epipolar_right_pin(new PinholeModel);
  epipolar(left_pin,  right_pin, *epipolar_left_pin, *epipolar_right_pin);

  // Get the input image crop regions, if any.
  BBox2i left_bbox, right_bbox;
  get_input_image_crops(left_bbox, right_bbox);

  // Shift the epipolar cameras to line up with the top left corner of the image and also
  //  get the pixel sizes of -L.tif and -R.tif.
  // - These camera model still represent the entire image as if no cropping occurred.
  resize_epipolar_cameras_to_fit(left_pin, right_pin,
                                 *(epipolar_left_pin.get()), *(epipolar_right_pin.get()),
                                 left_bbox, right_bbox, left_out_size, right_out_size);

  // The pinhole epipolar case is incompatible with adjustment files so they are not loaded.
  left_cam  = epipolar_left_pin;
  right_cam = epipolar_right_pin;
}

// Return the left transform used in alignment
StereoSessionPinhole::tx_type StereoSessionPinhole::tx_left() const {

  // A very annoying feature of epipolar alignment is that the cameras
  // returned in this mode already have alignment applied to them.
  // Then, the alignment need not happen in here, so we always return
  // tx_left_homography(), even though for epipolar alignment that
  // becomes the identity matrix instead of the expected alignment
  // transform.

  // It has to be that way since the cameras are aligned. Otherwise,
  // since triangulation is done with aligned cameras, if tx_left()
  // would return unaligned pixels, the results end up wrong.
  
  // That was likely a clever optimization, but it wreaks havoc with
  // usual conventions.  It also results in having two camera loaders,
  // namely camera_models() and get_unaligned_camera_models().

  // All this must be wiped and one must go back to the normal way of
  // doing things consistent with the other modes.  It would require
  // very careful understanding of the many camera loading functions
  // in this class, which have repeated logic, and how they deal with
  // alignment and crop windows. Lots of work.

  // So, for now leave things so that this class is consistent with
  // itself for epipolar alignment, even though that is not consistent
  // with the usual way of doing things.
  
  //if (stereo_settings().alignment_method != "epipolar")
  return StereoSession::tx_left_homography(); // see above

  // Comment out the logic below, per the text above.
  // TODO(oalexan1): Figure out if things can work without casting
  // away the const.
  //StereoSession::tx_type trans_left, trans_right;
  //((StereoSessionPinhole*)this)->pinhole_cam_trans(trans_left, trans_right);
  //return trans_left;
}

// Return the right transform used in alignment
StereoSessionPinhole::tx_type StereoSessionPinhole::tx_right() const {

  //if (stereo_settings().alignment_method != "epipolar")
  return StereoSession::tx_right_homography(); // see above

  // See the lengthy rant in tx_left() above.
  // TODO(oalexan1): Figure out if things can work without casting
  // away the const.
  //StereoSession::tx_type trans_left, trans_right;
  //((StereoSessionPinhole*)this)->pinhole_cam_trans(trans_left, trans_right);
  //return trans_right;
}

void StereoSessionPinhole::pinhole_cam_trans(tx_type & left_trans,
                                             tx_type & right_trans) {

  // Load the epipolar aligned camera models
  boost::shared_ptr<camera::CameraModel> left_aligned_model, right_aligned_model;

  // TODO(oalexan1): Models must be loaded by now, presumably. Then
  // pinhole_cam_trans can be const, and we don't to cast away the const in tx_left()
  // and tx_right() above.
  this->camera_models(left_aligned_model, right_aligned_model);
  
  boost::shared_ptr<camera::CameraModel> left_input_model, right_input_model;
  this->get_unaligned_camera_models(left_input_model, right_input_model);

  // Check the type, CAHV* type models are not supported!
  typedef vw::camera::PinholeModel PinModel;
  PinModel* left_in_ptr = dynamic_cast<PinModel*>(&(*left_input_model));
  if (!left_in_ptr)
    vw_throw(NoImplErr() << "Detected CAHV-type cameras. Use an alignment "
             << "method different than 'epipolar'.\n" );

  // Set up transform objects
  left_trans.reset (new PinholeCamTrans(*dynamic_cast<PinModel*>(&(*left_input_model)),
                                        *dynamic_cast<PinModel*>(&(*left_aligned_model))));
  right_trans.reset(new PinholeCamTrans(*dynamic_cast<PinModel*>(&(*right_input_model)),
                                        *dynamic_cast<PinModel*>(&(*right_aligned_model))));
}

} // end namespace asp

