// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file StereoSessionASTER.cc
///
#include <vw/Image/ImageMath.h>
#include <vw/Image/Manipulation.h>
#include <vw/Image/MaskViews.h>
#include <vw/Image/Transform.h>
#include <vw/Camera/CameraModel.h>

#include <asp/Core/AffineEpipolar.h>
#include <asp/Camera/LinescanASTERModel.h>
#include <asp/Sessions/StereoSessionASTER.h>


#include <iostream>
#include <string>
#include <vector>
#include <algorithm>

using namespace vw;
using namespace vw::camera;
using namespace asp;


namespace pt = boost::posix_time;
namespace fs = boost::filesystem;

namespace asp {

  // Get the RPC camera for a given ASTER camera. Respect the adjustments, if present.
  boost::shared_ptr<vw::camera::CameraModel>
  rpcModel(boost::shared_ptr<vw::camera::CameraModel> base_cam) {

    AdjustedCameraModel* adj_cam = dynamic_cast<AdjustedCameraModel*>(base_cam.get());

    if (adj_cam == NULL) {
      // Not an adjusted camera

      // Check if this is an ASTER camera
      ASTERCameraModel * aster_cam = dynamic_cast<ASTERCameraModel*>(base_cam.get());
      if (aster_cam == NULL)
        vw_throw(ArgumentErr() << "An ASTER camera model is expected in StereoSessionASTER.");
      
      // Return the RPC model
      return aster_cam->get_rpc_model();
    }

    // The case of an adjusted camera. Must pass the adjustments to
    // the RPC camera. This is very important, as the options
    // --left-image-crop-win and --right-image-crop-win change the
    // value of pixel_offset() below, and bundle adjustment may have
    // been done, which affects translation() and rotation().

    // Strip the adjustments
    boost::shared_ptr<vw::camera::CameraModel> unadj_cam;
    unadj_cam = adj_cam->unadjusted_model();
    
    // Sanity check
    ASTERCameraModel * aster_cam = dynamic_cast<ASTERCameraModel*>(unadj_cam.get());
    if (aster_cam == NULL)
      vw_throw(ArgumentErr() << "An ASTER camera model is expected in StereoSessionASTER.");
    
    // Get the RPC models
    boost::shared_ptr<vw::camera::CameraModel> rpc_cam = aster_cam->get_rpc_model();

    // Apply the adjustments and return
    return boost::shared_ptr<vw::camera::CameraModel>
      (new AdjustedCameraModel(rpc_cam, adj_cam->translation(), adj_cam->rotation(),
                               adj_cam->pixel_offset(), adj_cam->scale()));
    }  
  
  // This function will get the RPC models approximating the ASTER
  // models.  We want to use the RPC models for ip matching, as they
  // are way faster.  That does not affect the accuracy of the final
  // DEM, as interest points are only used to guess rough alignment
  // transforms and an initial search range.
  void StereoSessionASTER::rpc_camera_models(boost::shared_ptr<vw::camera::CameraModel> &cam1,
                                             boost::shared_ptr<vw::camera::CameraModel> &cam2){
    
    boost::shared_ptr<vw::camera::CameraModel> base_cam1, base_cam2;
    this->camera_models(base_cam1, base_cam2);
    
    cam1 = rpcModel(base_cam1);
    cam2 = rpcModel(base_cam2);
  }
  
  boost::shared_ptr<vw::camera::CameraModel>  StereoSessionASTER::load_camera_model
    (std::string const& image_file, std::string const& camera_file, Vector2 pixel_offset) const{

    return load_adjusted_model(m_camera_loader.load_ASTER_camera_model(camera_file),
                              image_file, camera_file, pixel_offset);
  }
} // End namespace asp
