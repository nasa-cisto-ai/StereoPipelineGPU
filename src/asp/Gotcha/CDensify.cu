#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <asp/Gotcha/CDensify.h>
#include <asp/Gotcha/ALSC.h>

#include <fstream>
#include <iostream>
#include <cmath>

using namespace std;
using namespace cv;

namespace gotcha {

CDensify::CDensify(){}

CDensify::CDensify(CDensifyParam paramDense, std::vector<CTiePt> const& vecTPs,
                   cv::Mat imgL, cv::Mat imgR,
                   cv::Mat input_dispX, cv::Mat input_dispY, cv::Mat Mask){
  setParameters(paramDense, vecTPs, imgL, imgR, input_dispX, input_dispY, Mask);
}

void CDensify::setParameters(CDensifyParam paramDense, std::vector<CTiePt> const& vecTPs,
                             cv::Mat imgL, cv::Mat imgR,
                             cv::Mat input_dispX, cv::Mat input_dispY, cv::Mat Mask){
  m_paramDense  = paramDense;
  m_vecTPs      = vecTPs;
  m_imgL        = imgL;
  m_imgR        = imgR;
  m_input_dispX = input_dispX;
  m_input_dispY = input_dispY;
  m_Mask        = Mask;
  
  // loadImages();
  // m_nCount = 0;
  // cout << m_paramDense.m_paramGotcha.m_paramALSC.m_fEigThr<< endl;
}


#if 0
void CDensify::loadImages(){
  string strImgL, strImgR;
  // get input images
  strImgL = m_paramDense.m_strImgL;
  strImgR = m_paramDense.m_strImgR;
  setImages(strImgL, strImgR);
}
#endif

#if 0
bool CDensify::loadTPForDensification(string strTPFile){    
    bool bRes = true;
    bRes = loadTP(strTPFile);
    return bRes;
}
#endif

int CDensify::performDensitification(cv::Mat & output_dispX, cv::Mat & output_dispY){

    ///////////////////////////////////////////
    // validate the input images
    ///////////////////////////////////////////
    if ( m_imgL.data == NULL || m_imgR.data == NULL)
        return CDensifyParam::FILE_IO_ERR;

#if 0
    // TP are now kept in memory the whole time, so no need to load them
    if (!loadTPForDensification(m_paramDense.m_strTPFile))
        return CDensifyParam::FILE_IO_ERR;
#endif
    
    if(m_paramDense.m_nProcType == CDensifyParam::GOTCHA){
        double start = getTickCount();
        // Generate integer-float seed points (nb. feature detection only gives float-float seed points)
        // Also, this is necessary to keep the original seed data, as initial ALSC will remove some seeds if selected.

        //Function below run initial ALSC on disparity based TPs. If use, need to change:
        //1. doGotcha(m_imgL, m_imgR, vecSeedTemp, m_paramDense.m_paramGotcha, m_vectpAdded)
        //2. m_vectpAdded.insert(m_vectpAdded.end(), vecSeedTemp.begin(), vecSeedTemp.end());
        //3. need some change in doPGotcha
        //IMARS still need initial ALSC
        vector<CTiePt> vecSeedTemp = getIntToFloatSeed(m_vecTPs);
        m_vecTPs.clear();
        for (int j = 0; j < (int)vecSeedTemp.size(); j++){
            CTiePt tpTemp = vecSeedTemp.at(j);
            m_vecTPs.push_back(tpTemp);
        }
        //IMARS still need initial ALSC



        //cout << "CASP-GO INFO: starting Gotcha" << endl;
        if (!doGotcha(m_imgL, m_imgR, m_vecTPs, m_paramDense.m_paramGotcha, m_vectpAdded))
            return CDensifyParam::GOTCHA_ERR;
        // remove TP with large y disparity
        // nb. sometimes large y disparity can be produced even from a rectified pair

        double dYLimit = 100;
        vector<CTiePt>::iterator iter;
        for (iter = m_vectpAdded.begin(); iter < m_vectpAdded.end(); ){
            double dy = iter->m_ptL.y - iter->m_ptR.y;
            dy = sqrt(dy*dy);
            if (dy > dYLimit)
                m_vectpAdded.erase(iter);
            else
                iter++;
        }

        m_vectpAdded.insert(m_vectpAdded.end(), m_vecTPs.begin(), m_vecTPs.end());

        double end = getTickCount();
        procTime = (end - start)/getTickFrequency();

        //cout << "CASP-GO INFO: making data products" << endl;

        makeDataProducts();

        if (!saveResult(output_dispX, output_dispY))
            return CDensifyParam::FILE_IO_ERR;

        // Do not save the log, this is hard to manage with multiple instances
        // running in parallel.
        //if (!saveLog())
        //    return CDensifyParam::FILE_IO_ERR;

    }
    else if (m_paramDense.m_nProcType == CDensifyParam::P_GOTCHA){
        double start = getTickCount();
        if (!doPGotcha(m_paramDense.m_paramGotcha.m_nNeiType))
            return CDensifyParam::P_GOTCHA_ERR;
        // remove TP with large y disparity

        double dYLimit = 100; //IMARS
        vector<CTiePt>::iterator iter;
        for (iter = m_vectpAdded.begin(); iter < m_vectpAdded.end(); ){
            double dy = iter->m_ptL.y - iter->m_ptR.y;
            dy = sqrt(dy*dy);
            if (dy > dYLimit)
                m_vectpAdded.erase(iter);
            else
                iter++;
        }

        m_vectpAdded.insert(m_vectpAdded.end(), m_vecTPs.begin(), m_vecTPs.end());

        double end = getTickCount();
        procTime = (end - start)/getTickFrequency();
        makeDataProducts();

        if (!saveResult(output_dispX, output_dispY))
            return CDensifyParam::FILE_IO_ERR;

        // Do not save the log, this is hard to manage with multiple instances
        // running in parallel.
        //if (!saveLog())
        //    return CDensifyParam::FILE_IO_ERR;
    }

    return CDensifyParam::NO_ERR;
}

vector<CTiePt> CDensify::getIntToFloatSeed(vector<CTiePt>& vecTPSrc) {
    vector<CTiePt> vecRes;

    int nLen =  vecTPSrc.size();
    for (int i = 0; i < nLen; i++){
        //get four neighbours
        CTiePt tp = vecTPSrc.at(i);
        Point2f ptL = tp.m_ptL;
        Point2f ptR = tp.m_ptR;
        float dX = 0, dY = 0; // xy offset to make integer seed

        dX = floor(ptL.x) - ptL.x;
        dY = floor(ptL.y) - ptL.y;
        if (dX == 0 && dY == 0){
            vecRes.push_back(tp);
            continue;
        }

        Point2f ptDelta(dX,dY);
        Point2f ptIntL = ptL + ptDelta; // ptIntL may be in float if ptL is came from the Rectified coordinate
        Point2f ptIntR = ptR + ptDelta;

        /////////////////////////////////////////////////////
        // collect 4 integer seed and validate
        /////////////////////////////////////////////////////
        CTiePt tpTemp = tp;
        vector<CTiePt> vectpSeeds;
        // pt1 (top-left)
        tpTemp.m_ptL = ptIntL;
        tpTemp.m_ptR = ptIntR;
        vectpSeeds.push_back(tpTemp);
        // pt2 (top-right)
        tpTemp.m_ptL = ptIntL + Point2f(1, 0);
        tpTemp.m_ptR = ptIntR + Point2f(1, 0);
        vectpSeeds.push_back(tpTemp);

        // pt3 (bottom-left)
        tpTemp.m_ptL = ptIntL + Point2f(0, 1);
        tpTemp.m_ptR = ptIntR + Point2f(0, 1);
        vectpSeeds.push_back(tpTemp);

        // pt 4 (bottom-right)    
        tpTemp.m_ptL = ptIntL + Point2f(1, 1);
        tpTemp.m_ptR = ptIntR + Point2f(1, 1);
        vectpSeeds.push_back(tpTemp);

        //apply ALSC to collect as new seed
        ALSC alsc(m_imgL, m_imgR,  m_paramDense.m_paramGotcha.m_paramALSC);
        alsc.performALSC(&vectpSeeds);
        vectpSeeds.clear();
        alsc.getRefinedTps(vectpSeeds); // hard-copy

        for (int j = 0; j < (int)vectpSeeds.size(); j++){
            CTiePt tpRef = vectpSeeds.at(j);
            if (tpRef.m_fSimVal != CTiePt::NOT_DEF){
                vecRes.push_back(tpRef);
            }
        }
    }

    return vecRes;
}

bool CDensify::saveProjLog(string strFile){

  return true;
#if 0
  std::cout << "Writing log " << strFile << std::endl;
  bool bRes = false;
  ofstream sfLog;
  sfLog.open(strFile.c_str(), ios::app | ios::out);
  string strProcType = getProcType();
  
  if (sfLog.is_open()){
    sfLog << "<Project I/O>" << endl;
    sfLog << "Input left image path: " << m_paramDense.m_strImgL << endl;
    sfLog << "Input right image path: " << m_paramDense.m_strImgR << endl;
    sfLog << "Processing type: " << strProcType << endl;
    
    sfLog << "Input x disparity map path: " << m_paramDense.m_strDispX << endl;
    sfLog << "Input y disparity map path: " << m_paramDense.m_strDispY << endl;
    sfLog << "Output tie-point file (from disparity) path: " << m_paramDense.m_strTPFile << endl;
    sfLog << "Input/output mask file path: " << m_paramDense.m_paramGotcha.m_strMask << endl;
    sfLog << "Output x disparity map path: " << m_paramDense.m_strUpdatedDispX << endl;
    sfLog << "Output y disparity map path: " << m_paramDense.m_strUpdatedDispY << endl;
    sfLog << endl;
    sfLog.close();
    bRes = true;
  }
  else
    return bRes = false;
  
  return bRes;
#endif
  
}

bool CDensify::saveLog(){
  return true;
#if 0
  string FILE_LOG = "-GLog.txt";

  string strFile = m_paramDense.m_strOutPath + FILE_LOG;
  bool bRes = false;
  
  // save parameter log
  //bRes = saveProjLog(strFile);
  bRes = saveGOTCHAParam(m_paramDense.m_paramGotcha, strFile);
  bRes = bRes && saveALSCParam(m_paramDense.m_paramGotcha.m_paramALSC, strFile);
  // save result log
  bRes = bRes && saveResLog(strFile);
  
  return bRes;
#endif
}

bool CDensify::saveResLog(string strFile){
  return true;
#if 0
    bool bRes = false;
    int nNumFinalTPs = nNumSeedTPs+m_vectpAdded.size();
    ofstream sfLog;
    sfLog.open(strFile.c_str(), ios::app | ios::out);

    if (sfLog.is_open()){
        sfLog << "<Processing results>" << endl;
        sfLog << "Processing method: " << m_paramDense.getProcessingType() << endl;
        sfLog << "Total number of seed TPs: " << nNumSeedTPs << endl;
        sfLog << "Total number of final TPs: " << nNumFinalTPs << endl;
        sfLog << "Total processing time(sec): " << procTime << endl;
        sfLog << endl;
        sfLog << endl;
        sfLog.close();
        bRes = true;
    }
    else
        return bRes = false;

    return bRes;
#endif
}

void CDensify::makeDataProducts(){

    // prepare output data product from the list of densified tiepoints    
    // make output products (disparity map x,y and sim)
    int nW = m_imgL.cols;
    int nH = m_imgL.rows;

    // clear&initialise output buffers
    m_matDisMapX = Mat::ones(nH, nW, CV_32FC1)*0.0;
    m_matDisMapY = Mat::ones(nH, nW, CV_32FC1)*0.0;
    m_matDisMapSim = Mat::ones(nH, nW, CV_32FC1)*-1;

    // fill the disparity map
    int nLen = m_vectpAdded.size();
    for (int i = 0 ; i < nLen; i++){
        CTiePt tp = m_vectpAdded.at(i);
        Point2f ptL = tp.m_ptL;
        Point2f ptR = tp.m_ptR;
        float fSim = tp.m_fSimVal;

        int x = (int) ptL.x;
        int y = (int) ptL.y;
        Rect_<int> rect(0,0,nW,nH);
        if (rect.contains(Point2i(x,y))) {
            m_matDisMapX.at<float>(y,x) = ptR.x - x;
            m_matDisMapY.at<float>(y,x) = ptR.y - y;
            m_matDisMapSim.at<float>(y,x) = fSim;
        }
    }        
}


bool CDensify::saveResult(cv::Mat & output_dispX, cv::Mat & output_dispY){

    bool bRes = true;

    //string strFile = m_paramDense.m_strTPFile;
    //bRes = saveTP(m_vectpAdded, strFile);
    
    //cout << "Writing results..." << endl;

    output_dispX = Mat::ones(m_matDisMapX.size(), CV_32FC1)*0.0;
    output_dispY = Mat::ones(m_matDisMapY.size(), CV_32FC1)*0.0;

    for (int i =0; i<output_dispX.rows; i++){
        for (int j=0; j<output_dispX.cols; j++){
            if (m_matDisMapX.at<float>(i,j)!= 0.0){ //-3.40282346639e+038
                float disp = m_matDisMapX.at<float>(i,j);
                output_dispX.at<float>(i,j)=disp;
            }
        }
    }

    for (int i =0; i<output_dispY.rows; i++){
        for (int j=0; j<output_dispY.cols; j++){
            if (m_matDisMapY.at<float>(i,j)!= 0.0){
                float disp = m_matDisMapY.at<float>(i,j);
                output_dispY.at<float>(i,j)=disp;
            }
        }
    }

    //Mat m_input_dispX = imread(m_paramDense.m_strDispX, CV_LOAD_IMAGE_ANYDEPTH);
    //Mat m_input_dispY = imread(m_paramDense.m_strDispY, CV_LOAD_IMAGE_ANYDEPTH);
    //std::cout << "Reading mask: " << m_paramDense.m_strMask << std::endl;
    //Mat Mask = imread(m_paramDense.m_strMask, CV_LOAD_IMAGE_ANYDEPTH);
    //Mask.convertTo(Mask, CV_8UC1);
    nNumSeedTPs = 0;

    if (m_input_dispX.depth()==2 && m_input_dispY.depth()==2){
        for (int i=0; i<m_input_dispX.rows; i++){
            for (int j=0; j<m_input_dispX.cols; j++){
                if (m_Mask.at<uchar>(i,j)==1){
                    m_input_dispX.at<ushort>(i,j)=65535;
                    m_input_dispY.at<ushort>(i,j)=65535;
                }
            }
        }
    }
    else if (m_input_dispX.depth()==5 && m_input_dispY.depth()==5){
        for (int i=0; i<m_input_dispX.rows; i++){
            for (int j=0; j<m_input_dispX.cols; j++){
                if (m_Mask.at<uchar>(i,j)==1){
                    m_input_dispX.at<float>(i,j)= 0.0;
                    m_input_dispY.at<float>(i,j)= 0.0;
                }
            }
        }
    }

    if (m_input_dispX.depth()==2 && m_input_dispY.depth()==2){
        for (int i=0; i<m_input_dispX.rows; i++){
            for (int j=0; j<m_input_dispX.cols; j++){
                if (m_input_dispX.at<ushort>(i,j)!=65535 && m_input_dispY.at<ushort>(i,j)!=65535)
                    nNumSeedTPs+=1;
            }
        }
    }
    else if (m_input_dispX.depth()==5 && m_input_dispY.depth()==5){
        for (int i=0; i<m_input_dispX.rows; i++){
            for (int j=0; j<m_input_dispX.cols; j++){
                if (m_input_dispX.at<float>(i,j)!= 0.0 && m_input_dispY.at<float>(i,j)!= 0.0)
                    nNumSeedTPs+=1;
            }
        }
    }


    //Recover values of original disp map
    if (m_input_dispX.depth()==2 && m_input_dispY.depth()==2){
        for (int i =0; i<output_dispX.rows; i++){
            for (int j=0; j<output_dispX.cols; j++){
                if (m_input_dispX.at<ushort>(i,j)!=65535)
                    output_dispX.at<ushort>(i,j)=m_input_dispX.at<ushort>(i,j);
                if (m_input_dispY.at<ushort>(i,j)!=65535)
                    output_dispY.at<ushort>(i,j)=m_input_dispY.at<ushort>(i,j);
            }
        }
    }
    else if (m_input_dispX.depth()==5 && m_input_dispY.depth()==5){
        for (int i =0; i<output_dispX.rows; i++){
            for (int j=0; j<output_dispX.cols; j++){
                if (m_input_dispX.at<float>(i,j)!= 0.0){  //IMARS
                    float disp = m_input_dispX.at<float>(i,j); //IMARS
                    output_dispX.at<float>(i,j)=disp; //IMARS
                }
                if (m_input_dispY.at<float>(i,j)!= 0.0){  //IMARS
                    float disp = m_input_dispY.at<float>(i,j);  //IMARS
                    output_dispY.at<float>(i,j)=disp; //IMARS
                }
            }
        }
    }
    //Cancelled in IMARS, since using sGotcha, we have whole map.

#if 0
    // The outputs are now passed out rather than saved to disk
    
    string strFileC1 = m_paramDense.m_strUpdatedDispX;
    bRes = bRes && saveMatrix(output_dispX, strFileC1);
    //imwrite(strFile, output_dispX);
    string strFileC2 = m_paramDense.m_strUpdatedDispY;
    bRes = bRes && saveMatrix(output_dispY, strFileC2);
    //imwrite(strFile, output_dispY);

    // Do not write this file as it is not used. If needed, such a
    // file must be written for every tile ASP processes.
    string strFileC3 = m_paramDense.m_strUpdatedDispSim;
    std::cout << "Saving matrix: " << strFileC3 << std::endl;
    bRes = bRes && saveMatrix(m_matDisMapSim, strFileC3);

    string strC1Tiff = "-c1_refined.tif";
    string strC2Tiff = "-c2_refined.tif";
    string strC3Tiff = "-uncertainty.tif";
    
    string strFileC1Tiff = m_paramDense.m_strOutPath + strC1Tiff;
    string strFileC2Tiff = m_paramDense.m_strOutPath + strC2Tiff;
    string strFileC3Tiff = m_paramDense.m_strOutPath + strC3Tiff;
    
    ostringstream strCmdGdalConversionC1;
    strCmdGdalConversionC1 << "gdal_translate " << strFileC1 << " -of GTiff " << strFileC1Tiff;
    std::cout << strCmdGdalConversionC1.str() << std::endl;
    system(strCmdGdalConversionC1.str().c_str());

    ostringstream strCmdGdalConversionC2;
    strCmdGdalConversionC2 << "gdal_translate " << strFileC2 << " -of GTiff " << strFileC2Tiff;
    std::cout << strCmdGdalConversionC2.str() << std::endl;
    system(strCmdGdalConversionC2.str().c_str());

    ostringstream strCmdGdalConversionC3;
    strCmdGdalConversionC3 << "gdal_translate " << strFileC3 << " -of GTiff " << strFileC3Tiff;
    std::cout << strCmdGdalConversionC3.str() << std::endl;
    system(strCmdGdalConversionC3.str().c_str());
#endif
    
    return bRes;
}

void CDensify::getNeighbour(const CTiePt tp, vector<CTiePt>& vecNeiTp, const int nNeiType, const Mat& matSim){
    //
    Point2f ptLeft = tp.m_ptL;
    Point2f ptRight = tp.m_ptR;

    if (nNeiType == CGOTCHAParam::NEI_DIFF){
        getDisffusedNei(vecNeiTp, tp, matSim);
    }
    /*if(nNeiType == CGOTCHAParam::NEI_X || nNeiType == CGOTCHAParam::NEI_Y || nNeiType == CGOTCHAParam::NEI_4 || nNeiType == CGOTCHAParam::NEI_8)*/
    else {
        if (nNeiType == CGOTCHAParam::NEI_4  || nNeiType == CGOTCHAParam::NEI_Y || nNeiType == CGOTCHAParam::NEI_8){
            CTiePt tp2;
            tp2.m_ptL = ptLeft + Point2f(0,1);
            tp2.m_ptR = ptRight + Point2f(0,1);
            vecNeiTp.push_back(tp2);
            tp2.m_ptL = ptLeft + Point2f(0,-1);
            tp2.m_ptR = ptRight + Point2f(0,-1);
            vecNeiTp.push_back(tp2);
        }

        if (nNeiType == CGOTCHAParam::NEI_X || nNeiType == CGOTCHAParam::NEI_4 || nNeiType == CGOTCHAParam::NEI_8){

            CTiePt tp;
            tp.m_ptL = ptLeft + Point2f(1,0);
            tp.m_ptR = ptRight + Point2f(1,0);
            vecNeiTp.push_back(tp);
            tp.m_ptL = ptLeft + Point2f(-1,0);
            tp.m_ptR = ptRight + Point2f(-1,0);
            vecNeiTp.push_back(tp);
        }

        if (nNeiType == CGOTCHAParam::NEI_8){
            CTiePt tp8;
            tp8.m_ptL = ptLeft + Point2f(-1,-1);
            tp8.m_ptR = ptRight + Point2f(-1,-1);
            vecNeiTp.push_back(tp8);
            tp8.m_ptL = ptLeft + Point2f(1,1);
            tp8.m_ptR = ptRight + Point2f(1,1);
            vecNeiTp.push_back(tp8);
            tp8.m_ptL = ptLeft + Point2f(-1,1);
            tp8.m_ptR = ptRight + Point2f(-1,1);
            vecNeiTp.push_back(tp8);
            tp8.m_ptL = ptLeft + Point2f(1,-1);
            tp8.m_ptR = ptRight + Point2f(1,-1);
            vecNeiTp.push_back(tp8);
        }
    }
}

void CDensify::getDisffusedNei(vector<CTiePt>& vecNeiTp, const CTiePt tp, const Mat& matSim){
    // estimate the growth
    // make a diffusion map
    int nSzDiff = m_paramDense.m_paramGotcha.m_paramALSC.m_nPatch;//12
    double pdDiffMap[nSzDiff*2+1][nSzDiff*2+1];
    int nW = m_imgL.cols;//m_imgL.getMaxX()+1;//m_imgL.getWidth();
    int nH = m_imgL.rows;//m_imgL.getMaxY()+1;//m_imgL.getHeight();
    Rect rectRegion(0,0,nW, nH);

    Point2f ptLeft = tp.m_ptL;
    Point2f ptRight = tp.m_ptR;
    for (int j = -nSzDiff ; j < nSzDiff + 1; j++){
        for (int i = -nSzDiff ; i < nSzDiff + 1; i++){
            double val = 0.f;
            int nX = (int)floor(ptLeft.x+i);
            int nY = (int)floor(ptLeft.y+j);
            if (rectRegion.contains(Point(nX, nY)) &&
                matSim.at<float>(nY, nX) > 0){
                // assume sim value has been already normalised
                val = 1.f - matSim.at<float>(nY, nX);// /m_paramDense.m_paramGotcha.m_paramALSC.m_fEigThr;
            }
            pdDiffMap[j+nSzDiff][i+nSzDiff] = val;
        }
    }

    // heat diffusion equation
    double dAlpha = m_paramDense.m_paramGotcha.m_fDiffCoef;// 0.05; // diffusion coefficient
    double dThr = m_paramDense.m_paramGotcha.m_fDiffThr;// 0.1;
    int nIter = m_paramDense.m_paramGotcha.m_nDiffIter;// 5;

    int nRow = nSzDiff*2+1;
    int nCol = nSzDiff*2+1;
    double pdDiffMapNext[nRow][nCol];
    double pdDiffMapCurrnet[nRow][nCol];

    for(int j = 0 ; j < nRow; j++){
        for(int i = 0; i < nCol; i++){
            pdDiffMapCurrnet[j][i] = pdDiffMap[j][i];
            pdDiffMapNext[j][i] = 0;
        }
    }

    // define neighbours
    for (int k = 0 ; k < nIter; k++){
        // get diffused
        double pdTemp[nRow][nCol];
        for (int j = 1; j < nRow-1; j++){
            for (int i = 1; i < nCol-1; i++){
                pdDiffMapNext[j][i] = pdDiffMapCurrnet[j][i] + dAlpha *
                                      (pdDiffMapCurrnet[j+1][i] + pdDiffMapCurrnet[j][i+1] +
                                       pdDiffMapCurrnet[j-1][i] + pdDiffMapCurrnet[j][i-1] - 4*pdDiffMapCurrnet[j][i]);
                pdTemp[j][i] = pdDiffMapNext[j][i];
            }
        }
        for(int j = 1 ; j < nRow-1; j++){
            for(int i = 1; i < nCol-1; i++){
                pdDiffMapCurrnet[j][i] = pdTemp[j][i];
            }
        }
    }

    int count = 0;

    for (int j = 1; j < nRow-1; j++){
        for (int i = 1; i < nCol-1; i++){
            if (pdDiffMapNext[j][i] > dThr) { //&& (i != nSzDiff && j != nSzDiff)){ // add as a neighbour
                if (i == nSzDiff && j == nSzDiff) continue;
                CTiePt tpNei;
                tpNei.m_ptL.x = ptLeft.x + i - nSzDiff;
                tpNei.m_ptL.y = ptLeft.y + j - nSzDiff;
                tpNei.m_ptR.x = ptRight.x + i - nSzDiff;
                tpNei.m_ptR.y = ptRight.y + j - nSzDiff;

                vecNeiTp.push_back(tpNei);

                count++;
            }
        }
    }
            // add minimum num of neighbours
            CTiePt tp2;
            tp2.m_ptL = ptLeft + Point2f(0,1);
            tp2.m_ptR = ptRight + Point2f(0,1);
            if (!isHavingTP(vecNeiTp, tp2))
                vecNeiTp.push_back(tp2);

            tp2.m_ptL = ptLeft + Point2f(0,-1);
            tp2.m_ptR = ptRight + Point2f(0,-1);
            if (!isHavingTP(vecNeiTp, tp2))
                vecNeiTp.push_back(tp2);

            //CTiePt tp3;
            tp2.m_ptL = ptLeft + Point2f(1,0);
            tp2.m_ptR = ptRight + Point2f(1,0);
            if (!isHavingTP(vecNeiTp, tp2))
                vecNeiTp.push_back(tp2);

            tp2.m_ptL = ptLeft + Point2f(-1,0);
            tp2.m_ptR = ptRight + Point2f(-1,0);
            if (!isHavingTP(vecNeiTp, tp2))
                vecNeiTp.push_back(tp2);
}

bool CDensify::isHavingTP(vector<CTiePt>& vecNeiTp, CTiePt tp){
    bool bRes = false;
    int nLen = vecNeiTp.size();
    for (int i = 0; i < nLen; i++){
        if (vecNeiTp.at(i) == tp){
            bRes = true;
            break;
        }
    }
    return bRes;
}

void CDensify::removeOutsideImage(vector<CTiePt>& vecNeiTp, const Rect_<float> rectTileL, const Rect_<float> rectImgR){
    vector<CTiePt>::iterator iter;

    for (iter = vecNeiTp.begin(); iter < vecNeiTp.end(); ){

        if (!rectTileL.contains(iter->m_ptL) || !rectImgR.contains(iter->m_ptR))
            vecNeiTp.erase(iter);
       else
            iter++;
    }

}

void CDensify::removePtInLUT(vector<CTiePt>& vecNeiTp, const vector<bool>& pLUT, const int nWidth){
    vector<CTiePt>::iterator iter;

    for (iter = vecNeiTp.begin(); iter < vecNeiTp.end(); ){        
        int nX = (int)floor(iter->m_ptL.x);
        int nY = (int)floor(iter->m_ptL.y);
        //int nIdx = nY * m_imgL.cols + nX;
        int nIdx = nY * nWidth + nX;

        if (pLUT[nIdx])
            vecNeiTp.erase(iter);
        else
            iter++;
    }
}

void CDensify::makeTiles(vector< Rect_<float> >& vecRectTiles, int nMin){

    vector< Rect_<float> >::iterator iter;
    for (iter = vecRectTiles.begin(); iter < vecRectTiles.end(); ){
        Rect_<float> rectParent = *iter;
        if (rectParent.width/2 > nMin && rectParent.height/2 > nMin){
            vecRectTiles.erase(iter);
            vector< Rect_<float> > vecChild;
            breakIntoSubRect(rectParent, vecChild);
            vecRectTiles.insert(vecRectTiles.end(), vecChild.begin(),  vecChild.end());
            iter = vecRectTiles.begin();
        }
        else
            iter++;
    }
}

void CDensify::breakIntoSubRect(Rect_<float> rectParent, vector< Rect_<float> >& vecRes){

    float fHalfW = rectParent.width/2.f;
    float fHalfH = rectParent.height/2.f;

    Point2f ptTopLeft[4];
    ptTopLeft[0] = Point2f(rectParent.x, rectParent.y);
    ptTopLeft[1] = Point2f(rectParent.x+fHalfW, rectParent.y);
    ptTopLeft[2] = Point2f(rectParent.x, rectParent.y+fHalfH);
    ptTopLeft[3] = Point2f(rectParent.x+fHalfW, rectParent.y+fHalfH);

    Size sz(fHalfW, fHalfH);
    Rect_<float> rect(ptTopLeft[0], sz);
    vecRes.push_back(rect);

    sz = Size(rectParent.x+rectParent.width - ptTopLeft[1].x, fHalfH);
    rect = Rect(ptTopLeft[1], sz);
    vecRes.push_back(rect);

    sz = Size(fHalfW, rectParent.y+rectParent.height - ptTopLeft[2].y);
    rect = Rect(ptTopLeft[2], sz);
    vecRes.push_back(rect);

    sz = Size(rectParent.x+rectParent.width - ptTopLeft[3].x, rectParent.y+rectParent.height - ptTopLeft[3].y);
    rect = Rect(ptTopLeft[3], sz);
    vecRes.push_back(rect);

}


bool CDensify::doGotcha(const Mat& matImgL, const Mat& matImgR, vector<CTiePt>& vectpSeeds,
                        const CGOTCHAParam& paramGotcha, vector<CTiePt>& vectpAdded){


    bool bRes = true;
    ////////////////////////////////////////////
    // parameter preparation
    // cout << "CASP-GO INFO: initialisaing similarity map" << endl;
    Mat matSimMap = Mat::ones(matImgL.rows, matImgL.cols, CV_32FC1); // 2D similarity map for diffusion
    matSimMap = matSimMap*-1;

    // cout << "DEBUG: matImgL has statistics: rows " << matImgL.rows << "; cols: " << matImgL.cols << "; channels: " << matImgL.channels() << "; bitdepth code: " << matImgL.depth() << endl;

    Size szImgL(matImgL.cols, matImgL.rows);
    // cout << "CASP-GO INFO: initialising pixel LUT" << endl;

    // IMARS bool pLUT[szImgL.area()]; // if true it indicates the pixel has already processed
    vector<bool> pLUT; //IMARS

    // initialise
    for (int i = 0; i< szImgL.area(); i++){
        bool bpLUT = false; //IMARS
        pLUT.push_back(bpLUT); //IMARS
    } //IMARS

    vector< Rect_<float> > vecRectTiles;
    vecRectTiles.push_back(Rect(0., 0., matImgL.cols, matImgL.rows));

    // cout << "CASP-GO INFO: Making tiles" << endl;
    makeTiles(vecRectTiles, paramGotcha.m_nMinTile);

    if (paramGotcha.m_bNeedInitALSC){
        // cout << "CASP-GO INFO: Running initial ALSC refinement" << endl;
        ALSC alsc(matImgL, matImgR, paramGotcha.m_paramALSC);
        alsc.performALSC(&vectpSeeds);
        vectpSeeds.clear();
        alsc.getRefinedTps(vectpSeeds); // hard-copy
    }

    // cout << "CASP-GO INFO: initialise similarity map with seed points" << endl;
    // initialise sim map with seedpoitns
    for (int i = 0; i < (int)vectpSeeds.size(); i++){
        int nX =  (int)floor(vectpSeeds.at(i).m_ptL.x);
        int nY =  (int)floor(vectpSeeds.at(i).m_ptL.y);
        matSimMap.at<float>(nY, nX) = vectpSeeds.at(i).m_fSimVal;

        int nIdx = nY*szImgL.width + nX;
        pLUT[nIdx] = true;

        //cout << "DEBUG: pLUT change successfully to " << pLUT[nIdx] << endl;
    }

    // cout << "CASP-GO INFO: apply mask to remove area that no densification is required." << endl;
    // apply mask, remove area where no densification is required
    //std::cout << "Reading mask: " << paramGotcha.m_strMask << std::endl;
    //Mat Mask = imread(paramGotcha.m_strMask, CV_LOAD_IMAGE_ANYDEPTH);
    for (int i=0; i<m_Mask.rows; i++){
        for (int j=0; j<m_Mask.cols; j++){
            if (m_Mask.at<uchar>(i,j)==0){
                int nIdx = i*m_Mask.cols + j;
                pLUT[nIdx] = true;
                //cout << "DEBUG: pLUT change successfully to " << pLUT[nIdx] << endl;
            }
        }
    }

    /////////////////////////
    vectpAdded.clear();
    //cout << "CASP-GO INFO: Desifying disparity... ..." << endl;

    for (int i = 0 ; i < (int)vecRectTiles.size(); i++){
          vector<CTiePt> vecRes;
          bRes = bRes && doTileGotcha(matImgL, matImgR, vectpSeeds, paramGotcha, vecRes, vecRectTiles.at(i), matSimMap, pLUT);
          // collect result
          vectpAdded.insert(vectpAdded.end(), vecRes.begin(), vecRes.end());

          // debug
          //cout << "Tile " << i << " has been processed: " << vecRes.size() << " points are added" << endl;
    }

    return bRes;
}


bool CDensify::doTileGotcha(const Mat& matImgL, const Mat& matImgR, const
                            vector<CTiePt>& vectpSeeds,
                            const CGOTCHAParam& paramGotcha, vector<CTiePt>& vectpAdded,
                            const Rect_<float> rectTileL, Mat& matSimMap, vector<bool>& pLUT){

    vector<CTiePt> vectpSeedTPs; //= vectpSeeds;                // need this hard copy for sorting

    Size szImgL(matImgL.cols, matImgL.rows);
    Rect_<float> rectImgR (0, 0, matImgR.cols, matImgR.rows);
    vectpAdded.clear(); //clear output tp list

    // set indicator buffer
    for (int i = 0 ; i < (int)vectpSeeds.size(); i++){
        CTiePt tp = vectpSeeds.at(i);

        if (rectTileL.contains(tp.m_ptL)){
            vectpSeedTPs.push_back(tp);
        }

    }
    //std::cout << "Reading mask: " << paramGotcha.m_strMask << std::endl;
    //Mat Mask = imread(paramGotcha.m_strMask, CV_LOAD_IMAGE_ANYDEPTH);
    Mat imgL = matImgL;
    Mat imgR = matImgR;
    imgL.convertTo(imgL, CV_8UC1);
    imgR.convertTo(imgR, CV_8UC1);

    unsigned int nGapSize = 0;
    for (int i=0; i<m_Mask.rows; i++){
        for (int j=0; j< m_Mask.cols; j++){
          if (m_Mask.at<uchar>(i,j)==1 &&
              imgL.at<uchar>(i,j)!=0 &&imgR.at<uchar>(i,j)!=0)
                nGapSize+=1;
        }
    }

    //sort(vectpSeedTPs.begin(), vectpSeedTPs.end(), compareTP); // sorted in ascending order
    /////////////////////////////////////////////////////////////////////
    // stereo region growing
    //int nCount = 0; // for debug
#if 0
      // Hide the verbose messages
      cout << "[--------------------]  0% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
#endif
      
    while (vectpSeedTPs.size() > 0) {
        // get a point from seed
        CTiePt tp = vectpSeedTPs.at(0);

//        mvectpAdded.push_back(tp);
        vectpSeedTPs.erase(vectpSeedTPs.begin());
        vector<CTiePt> vecNeiTp;        
        getNeighbour(tp, vecNeiTp, paramGotcha.m_nNeiType, matSimMap);
        removeOutsideImage(vecNeiTp, rectTileL, rectImgR);
        removePtInLUT(vecNeiTp, pLUT, matImgL.cols);

        //ALSC
        if ((int)vecNeiTp.size() > 0){            
            // get affine data from a seed
            float pfData[6] = {0, 0, 0, 0, 0, 0};
            for (int k = 0; k < 4; k++)
                pfData[k] = tp.m_pfAffine[k];
            pfData[4] = tp.m_ptOffset.x;
            pfData[5] = tp.m_ptOffset.y;

            ALSC alsc(matImgL, matImgR, paramGotcha.m_paramALSC);

            alsc.performALSC(&vecNeiTp, (float*) pfData);
            const vector<CTiePt>* pvecRefTPtemp = alsc.getRefinedTps();

            int nLen = pvecRefTPtemp->size();
            if( nLen > 0){
                // append survived neighbours to the seed point list and the seed LUT
                // vector<CRefinedTP>::iterator iterNei;
                for (int i = 0 ; i < nLen; i++){
                    CTiePt tpNei = pvecRefTPtemp->at(i);

                    int nXnei = (int)floor(tpNei.m_ptL.x);
                    int nYnei = (int)floor(tpNei.m_ptL.y);
                    int nIdxNei = nYnei*szImgL.width + nXnei;

                    matSimMap.at<float>(nYnei,nXnei) = tpNei.m_fSimVal;                    
                    pLUT[nIdxNei] = true;

                    vectpSeedTPs.push_back(tpNei);
                    vectpAdded.push_back(tpNei);
                }

                //sort(vectpSeedTPs.begin(), vectpSeedTPs.end(), compareTP);
            } 
        }
#if 0
        // Hide the verbose messages
        if (vectpAdded.size()>0){
        if (vectpAdded.size() >= (nGapSize)/20 && vectpAdded.size() < 2*(nGapSize)/20)
            cout << "[*-------------------]  5% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 2*(nGapSize)/20 && vectpAdded.size() < 3*(nGapSize)/20)
            cout << "[**------------------] 10% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 3*(nGapSize)/20 && vectpAdded.size() < 4*(nGapSize)/20)
            cout << "[***-----------------] 15% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 4*(nGapSize)/20 && vectpAdded.size() < 5*(nGapSize)/20)
            cout << "[****----------------] 20% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 5*(nGapSize)/20 && vectpAdded.size() < 6*(nGapSize)/20)
            cout << "[*****---------------] 25% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 6*(nGapSize)/20 && vectpAdded.size() < 7*(nGapSize)/20)
            cout << "[******--------------] 30% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 7*(nGapSize)/20 && vectpAdded.size() < 8*(nGapSize)/20)
            cout << "[*******-------------] 35% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 8*(nGapSize)/20 && vectpAdded.size() < 9*(nGapSize)/20)
            cout << "[********------------] 40% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 9*(nGapSize)/20 && vectpAdded.size() < 10*(nGapSize)/20)
            cout << "[*********-----------] 45% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 10*(nGapSize)/20 && vectpAdded.size() < 11*(nGapSize)/20)
            cout << "[**********----------] 50% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 11*(nGapSize)/20 && vectpAdded.size() < 12*(nGapSize)/20)
            cout << "[***********---------] 55% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 12*(nGapSize)/20 && vectpAdded.size() < 13*(nGapSize)/20)
            cout << "[************--------] 60% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 13*(nGapSize)/20 && vectpAdded.size() < 14*(nGapSize)/20)
            cout << "[*************-------] 65% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 14*(nGapSize)/20 && vectpAdded.size() < 15*(nGapSize)/20)
            cout << "[**************------] 70% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 15*(nGapSize)/20 && vectpAdded.size() < 16*(nGapSize)/20)
            cout << "[***************-----] 75% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 16*(nGapSize)/20 && vectpAdded.size() < 17*(nGapSize)/20)
            cout << "[****************----] 80% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 17*(nGapSize)/20 && vectpAdded.size() < 18*(nGapSize)/20)
            cout << "[*****************---] 85% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 18*(nGapSize)/20 && vectpAdded.size() < 19*(nGapSize)/20)
            cout << "[******************--] 90% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() >= 19*(nGapSize)/20 && vectpAdded.size() < 20*(nGapSize)/20)
            cout << "[*******************-] 95% OF THE GAP AREA HAS BEEN DENSIFIED\r" << std::flush;
        else if (vectpAdded.size() == nGapSize)
          cout << "[********************]100% OF THE GAP AREA HAS BEEN DENSIFIED";
        }
#endif
    }
#if 0
    cout << endl;
#endif
    
    return true;
}

bool CDensify::doPGotcha(int nNeiType){

    // pyramid construction
    int nTotLev = getTotPyramidLev(m_paramDense.m_paramGotcha.m_paramALSC.m_nPatch);
    vector<Mat> vecImgPyL;
    vector<Mat> vecImgPyR;
    buildPyramid(m_imgL, vecImgPyL, nTotLev);
    buildPyramid(m_imgR, vecImgPyR, nTotLev);

    vector<CTiePt> vecOrgSeedClone = getIntToFloatSeed(m_vecTPs);
    if (m_paramDense.m_paramGotcha.m_bNeedInitALSC){
        ALSC alsc(m_imgL, m_imgR, m_paramDense.m_paramGotcha.m_paramALSC);
        alsc.performALSC(&vecOrgSeedClone);
        vecOrgSeedClone.clear();
        alsc.getRefinedTps(vecOrgSeedClone); // hard-copy
    }

    CGOTCHAParam paramG = m_paramDense.m_paramGotcha;
    paramG.m_nNeiType = nNeiType;
    paramG.m_bNeedInitALSC = true;

    float fTempLim = m_paramDense.m_paramGotcha.m_paramALSC.m_fEigThr;
    int nTempSz = m_paramDense.m_paramGotcha.m_paramALSC.m_nPatch;
    float fMinRate = 0.3f;
    float fMinRatePatch = 0.5f;

    vector<CTiePt> vecAddedTemp; // addition from previous level
    bool bRes = true;
    for (int i = nTotLev-1; i >= 0; i--){

        // prepare new processing param at level i
        paramG.m_paramALSC.m_nPatch = (int)(nTempSz * (1.f - fMinRatePatch * i/(nTotLev-1)));
        paramG.m_paramALSC.m_fEigThr = fTempLim * (1.f + fMinRate * i/(nTotLev-1));

        // prepare seed for current level
        vector<CTiePt> vecSeedTemp;
        float fDenom = pow(2.f, i);
        int nLen = vecOrgSeedClone.size();
        for (int j = 0; j < nLen; j++){
            CTiePt tp;
            tp = vecOrgSeedClone.at(j);
            tp.m_ptL.x = tp.m_ptL.x / fDenom;
            tp.m_ptL.y = tp.m_ptL.y / fDenom;
            tp.m_ptR.x = tp.m_ptR.x / fDenom;
            tp.m_ptR.y = tp.m_ptR.y / fDenom;
            for (int k = 0; k < 4 ; k++)
                tp.m_pfAffine[k] = 0.;
            tp.m_ptOffset.x = tp.m_ptOffset.x / fDenom;
            tp.m_ptOffset.y = tp.m_ptOffset.y / fDenom;
            vecSeedTemp.push_back(tp);
        }

        // added point propagated points
        nLen = vecAddedTemp.size();
        for (int j = 0; j < nLen; j++){
            vecAddedTemp.at(j).m_ptL.x *= 2.f;
            vecAddedTemp.at(j).m_ptL.y *= 2.f;
            vecAddedTemp.at(j).m_ptR.x *= 2.f;
            vecAddedTemp.at(j).m_ptR.y *= 2.f;
            for (int k = 0; k < 4 ; k++)
                vecAddedTemp.at(j).m_pfAffine[k] = 0.;
            vecAddedTemp.at(j).m_ptOffset.x *= 2.f;
            vecAddedTemp.at(j).m_ptOffset.y *= 2.f;
        }

        if (nLen > 0){
            ALSC alsc(vecImgPyL.at(i), vecImgPyR.at(i), paramG.m_paramALSC);
            alsc.performALSC(&vecAddedTemp);
            vecAddedTemp.clear();
            alsc.getRefinedTps(vecAddedTemp); // hard-copy
        }

        vector<CTiePt> seedIn;
        seedIn.insert(seedIn.end(), vecSeedTemp.begin(), vecSeedTemp.end() );
        seedIn.insert(seedIn.end(), vecAddedTemp.begin(), vecAddedTemp.end());
        vecSeedTemp.clear();

        // debug
        //cout << "# seed pts at the " << i << " level" << ": " << seedIn.size() << endl;

        vector<CTiePt> vecResTemp;
        bRes = bRes && doGotcha(vecImgPyL.at(i), vecImgPyR.at(i), seedIn, paramG, vecResTemp);
        vecAddedTemp.insert(vecAddedTemp.end(), vecResTemp.begin(), vecResTemp.end());

        // debug
        // cout << "# added pts at the " << i << " level" << ": " << vecResTemp.size() << endl;
    }

    m_vectpAdded.clear();
    m_vectpAdded.insert(m_vectpAdded.end(), vecOrgSeedClone.begin(), vecOrgSeedClone.end());
//    m_vectpAdded.insert(m_vectpAdded.end(), m_vecTPs.begin(), m_vecTPs.end());
    m_vectpAdded.insert(m_vectpAdded.end(), vecAddedTemp.begin(), vecAddedTemp.end());

    return bRes;
}

int CDensify::getTotPyramidLev(int nszPatch) {
    int nRes = 0;

    Size szImgL(m_imgL.cols, m_imgL.rows);
    Size szImgR(m_imgR.cols, m_imgR.rows);
    int nW = 0, nH = 0;

    // get smallest size of width
    nW = min(szImgL.width, szImgR.width);
    nH = min(szImgL.height, szImgR.height);

    int nMinSz = 4*(nszPatch * 2 + 1);
    Size szMin(nMinSz,nMinSz);

    // estimate the minimum of the largest levels
    while(nW > szMin.width || nH > szMin.height){
        nW /= 2;
        nH /= 2;
        nRes++;
    }

    return nRes;
}

} // end namespace gotcha
