#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file Metadata.cc
///

/* ------------------------------------------------------------------
 * MOCMetadata.cc
 *
 * General functions for reading MOC ephemeris information provided by
 * Malin Space Science System.  This file contains funcitons for reading
 * the following two types of data file:
 *
 * 1. description.tab files that contain tabulated MOC ephemeris information.
 *    This information is essentially what would be available in the PDS header.
 *
 * 2. SUP files that contain detailed information about camera position, velocity,
 *    pose, focal length, pixel summing, and image size for a given MOC image.
 *    There is also a function for generating a VRML model that depicts the
 *    location of the MGS orbiting Mars at the position and pose corresponding
 *    to two MOC-NA image shots.
 */
#include <asp/Sessions/MOC/Metadata.h>
#include <asp/Sessions/MOC/Ephemeris.h>
#include <asp/SpiceIO/SpiceUtilities.h>
#include <asp/SpiceIO/TabulatedDataReader.h>

// VisionWorkbench
#include <vw/Core/Log.h>
#include <vw/Camera/OrbitingPushbroomModel.h>

// Boost
#include <boost/algorithm/string.hpp>

// STL
#include <vector>
#include <string>
#include <iostream>                        // for WriteVizSiteFrame
#include <fstream>                         // for WriteVizSiteFrame

using namespace std;
using namespace vw;
// Circumference of Mars in meters:
static const double kMarsCircumference = 21344.0e6; // km
static const double kMarsRadius = 3397.0e3;             // km, equatorial (polar is 3395)


/* ----------------------------------------------------------------------
 *               MOCImageMetadata Class Methods
 * ----------------------------------------------------------------------*/

// Parse the MOC image number from the filename
MOCImageMetadata::MOCImageMetadata(std::string const& filename) {
  m_filename = filename;
  m_focal_length = 3.437;
  m_along_scan_pixel_size = 13e-6;
  m_across_scan_pixel_size = 13e-6;
  m_start_sample = -9999;

  // Change to upper case
  std::string name = boost::to_upper_copy(filename);

  // Remove any file suffix
  std::vector<string> suffix_split;
  boost::split( suffix_split, name, boost::is_any_of(".") );

  // Erase all seperators
  std::vector<string> vec;
  boost::split( vec, suffix_split[0], boost::is_any_of("-/") );
  if (vec.size() == 2) {
    string first = vec[0];
    string last = vec[1];

    // Fix discrepancy between FHGA and FHA
    if (first == "FHGA")
      first = "FHA";

    // Insert extra 0 if needed
    if (first.length() < 3)
      first.insert(1,"0");

    while (last.length() < 5)
      last.insert(0,"0");

    m_moc_identifier = (first + "/" + last);
  } else if (vec.size() == 1 && vec[0].size() > 5){
    int len = vec[0].size();
    m_moc_identifier = vec[0].substr(0,len-5) + "/" + vec[0].substr(len-5, len-1);
  } else {
    // In this case we assume that the
    std::cout << "WARNING: failed to parse the filename \"" << filename << "\"."
              << "Data lookup in \n";
    m_moc_identifier = filename;
  }
}


/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
 *             GENERATE CAMERA MODEL FROM FREE PARAMETERS
 * - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
vw::camera::CameraModel* MOCImageMetadata::camera_model() {

  int start_sample;
  if (m_start_sample == -9999) {
    std::cout << "WARNING: SETTING START_SAMPLE TO -1024.\n";
    start_sample = int(-1024/m_crosstrack_summing);
  } else {
    start_sample = int(m_start_sample/m_crosstrack_summing);
  }

  // Use the values that were obtained from the *.sup file to program
  // the camera model parameters.
  return new vw::camera::OrbitingPushbroomModel( rows(), // number of lines
                                                 cols(), // sampels per line
                                                 start_sample,
                                                 m_focal_length,
                                                 m_along_scan_pixel_size*m_downtrack_summing,
                                                 m_across_scan_pixel_size*m_crosstrack_summing,
                                                 scan_duration() / rows(), // line integration time
                                                 m_t0_quat, m_dt_quat,
                                                 m_t0_ephem, m_dt_ephem,
                                                 Vector3(0,0,1),  // pointing_vect
                                                 Vector3(0,1,0),  // u_vec
                                                 m_quat,   // camera poses
                                                 m_ephem); // camera positions
}

// Load satellite telemetry directly using SPICE.
void MOCImageMetadata::read_spice_data() {
  m_t0_ephem = -1.25;   m_dt_ephem = 0.25;
  m_t0_quat = -1.25;    m_dt_quat = 0.25;

  // For debugging:
  //   cout << "Scan duration " << scan_duration() << "\n";
  //   cout << "Range [" << ephemeris_time + m_t0_ephem << ", " << ephemeris_time + scan_duration() - m_t0_ephem << ", " << m_dt_ephem << "] \n";

  MOC_state(ephemeris_time() + m_t0_ephem,
            ephemeris_time() + scan_duration() - m_t0_ephem,
            m_dt_ephem, m_ephem, m_ephem_rate, m_quat);
}

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
 *              MOC DESCRIPTION.TAB FILE MANIPULATION
 *                    AND VIZ SITE FRAME I/O
 * - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

// --------------------------------------------------------
//                Ephemeris entry data
//            (one line of description.tab
// --------------------------------------------------------
/* Byte indices for fields in the MOC description.tab file */
enum { EPHEM_IMAGE_WIDTH = 6,           // pixels
       EPHEM_IMAGE_HEIGHT = 7,          // pixels
       EPHEM_CROSSTRACK_SUMMING = 8,    // unitless
       EPHEM_DOWNTRACK_SUMMING = 9,     // unitless
       EPHEM_ASPECT_RATIO = 11,         // unitless
       EPHEM_INCIDENCE_ANGLE = 12,      // degrees
       EPHEM_LONGITUDE = 15,            // degrees
       EPHEM_LATITUDE = 16,             // degrees
       EPHEM_SCLK_TIME = 27,            // system clock ticks
       EPHEM_ACTUAL_WIDTH = 34,         // kilometers
       EPHEM_ACTUAL_HEIGHT = 35,        // kilometers
       EPHEM_IMAGE_FLIPPED = 39,        // 'Y' or 'N'
       EPHEM_NORTH_ANGLE = 40,          // degrees
       EPHEM_SKEW_ANGLE = 49,           // degrees
       EPHEM_LINE_INTEGRATION_TIME = 29,// seconds
       EPHEM_ORBIT_NUMBER = 52          // orbits
};


void MOCImageMetadata::read_tabulated_description(std::string const& filename) {
  // Read in the information found in the description.tab file.
  try {
    TabulatedDataReader ephemeris_reader(filename, ",");
    vector<string> imagetab;
    cout << "\tFinding entries for " << m_moc_identifier << ".\n";
    if (!ephemeris_reader.find_line_with_text(m_moc_identifier, imagetab)) {
      cout << "Error: could not find an entry that matched " << m_moc_identifier << " in the description file " << filename << "\nExiting.\n\n";
      exit(0);
    }
    parse_ephemeris_entry(imagetab);
    ephemeris_reader.close();
  } catch (IOErr &e) {
    cout << "Failed to open the description file: " << filename << "\n\t";
    cout << e.what() << "\n";
    exit(0);
  }
}

void MOCImageMetadata::parse_ephemeris_entry(std::vector<std::string> ephemerisData)
{
  sscanf(ephemerisData[EPHEM_LONGITUDE].c_str(), "%lf", &m_longitude);          // Image center longitude
  sscanf(ephemerisData[EPHEM_LATITUDE].c_str(), "%lf", &m_latitude);              // Image center latitude
  sscanf(ephemerisData[EPHEM_IMAGE_WIDTH].c_str(), "%lf", &m_width_pixels);     // Image width in pixels
  sscanf(ephemerisData[EPHEM_IMAGE_HEIGHT].c_str(), "%lf", &m_height_pixels);   // Image height in pixels
  sscanf(ephemerisData[EPHEM_CROSSTRACK_SUMMING].c_str(), "%lf", &m_crosstrack_summing);
  sscanf(ephemerisData[EPHEM_DOWNTRACK_SUMMING].c_str(), "%lf", &m_downtrack_summing);
  sscanf(ephemerisData[EPHEM_ASPECT_RATIO].c_str(), "%lf", &m_aspect);
  sscanf(ephemerisData[EPHEM_ACTUAL_WIDTH].c_str(), "%lf", &m_width_meters); // Image width in kilometers
  sscanf(ephemerisData[EPHEM_ACTUAL_HEIGHT].c_str(), "%lf", &m_height_meters); // Image height in kilometers
  sscanf(ephemerisData[EPHEM_INCIDENCE_ANGLE].c_str(), "%lf", &m_angle);           // Incidence angle in degrees
  sscanf(ephemerisData[EPHEM_NORTH_ANGLE].c_str(), "%lf", &m_north_angle);
  sscanf(ephemerisData[EPHEM_SKEW_ANGLE].c_str(), "%lf", &m_skew_angle);
  sscanf(ephemerisData[EPHEM_LINE_INTEGRATION_TIME].c_str(), "%lf", &m_line_integration_time);  // Line integration time
  sscanf(ephemerisData[EPHEM_ORBIT_NUMBER].c_str(), "%d", &m_orbit_number);

  m_sclk_time = ephemerisData[EPHEM_SCLK_TIME];
  m_sclk_time.erase(0, 1);
  m_sclk_time.erase(m_sclk_time.size() - 1, 1);
  m_flip = (*(ephemerisData[EPHEM_IMAGE_FLIPPED].c_str()) == 'F');

  m_worldToLocal = GenWorldToLocalTransform(m_latitude, m_longitude, kMarsRadius);
  m_aspect = 1.0 / m_aspect;
  m_width_meters = m_width_meters * 1000.0;        // Image width in meters
  m_height_meters = m_height_meters * 1000.0;      // Image height in meters
  m_angle = m_angle * M_PI / 180;          // Incidence angle in radians
  m_north_angle = m_north_angle * M_PI / 180;
  m_skew_angle = m_skew_angle * M_PI / 180;
  m_line_integration_time /= 1000.0;              // Convert from ms to secs
}


// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
//                  MOC SUP FILE MANIPULATION
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
//
// Read a MOC *.sup data file provided by Malin Space Science Systems
// This file is used to initialize a pushbroom camera model.
void MOCImageMetadata::read_ephemeris_supplement(std::string const& filename) {

  SupplementaryEphemerisParser parser(filename);

  try {

    // First, gather the basic intrinsic camera prameters
    //
    // Note that these values may be overridden by the values in the
    // description.tab file (though they should be identical values).
    //
    m_crosstrack_summing = parser.read_double("PIXEL_SUMMING");
    m_downtrack_summing = parser.read_double("PIXEL_SUMMING");
    //    double scan_duration = parser.read_double("SCAN_DURATION");

    double downloaded_lines = parser.read_double("DOWNLOADED_LINES");
    m_height_pixels = downloaded_lines/m_downtrack_summing;

    double focal_length = parser.read_double("FOCAL_LENGTH") / 1000.0;
    if (focal_length != m_focal_length) { std::cout << "WARNING: File focal length differs from standard MOC focal length ( " << focal_length << "\n"; }

    double across_scan_pixel_size = parser.read_double("ACROSS_SCAN_PIXEL_SIZE") / 1.0e6;
    double along_scan_pixel_size = parser.read_double("ALONG_SCAN_PIXEL_SIZE") / 1.0e6;
    if (across_scan_pixel_size != m_across_scan_pixel_size) { std::cout << "WARNING: File across scan pixel size differs from standard MOC focal length.\n"; }
    if (along_scan_pixel_size != m_along_scan_pixel_size) { std::cout << "WARNING: File along scan pixel size differs from standard MOC focal length.\n"; }

    double downloaded_samples = parser.read_double("DOWNLOADED_SAMPLES");
    m_width_pixels = downloaded_samples/m_crosstrack_summing;

    // This is really the only piece of information we need from the ephemeris file.
    // (This does noet appear in the description.tab file)
    m_start_sample = parser.read_double("START_SAMPLE");

    /* Read the full ephemeris information into a Nx3 matrix*/
    double n_ephem = parser.read_double("N_EPHEM");
    m_t0_ephem = parser.read_double("T0_EPHEM");
    m_dt_ephem = parser.read_double("DT_EPHEM");
    m_ephem = parser.read_vector3s("EPHEM", (int)n_ephem, 3);
    m_ephem_rate = parser.read_vector3s("EPHEM_RATE", (int)n_ephem, 3);

    /* Next, read in the time serios of data regarding orientation */
    double n_quat = parser.read_double("NUM_QUAT");
    m_t0_quat = parser.read_double("T0_QUAT");
    m_dt_quat = parser.read_double("DT_QUAT");
    m_quat = parser.read_quaternions("QUATERNIONS", (int)n_quat, 4);

    /* For IRG generated SUP files, the quaternion already takes into account the minute
     * rotation between the MOC frame of reference and the MGS S/C frame of reference.
     *
     * However, MSSS quaternions do not (we think).
     *
     * The rotation matrix below could be used to correct the quaternions from MGS frame
     * to MOC-NA frame.
     *
     * This matrix is built from a set of Euler angles that were obtained from Malin
     * Space Science Systems.  Refer to docs/MGS_to_MOC_alignment.txt for a full explanation
     * of how this matrix is built.
     */
    //   Matrix<double> R_moc(3,3);
    //   R_moc(0,0) = 1.00000;   R_moc(0,1) = -0.0031;  R_moc(0,2) = 0.0002;
    //   R_moc(1,0) = 0.0031;    R_moc(1,1) = 1.0000;   R_moc(1,2) = 0.0012;
    //   R_moc(2,0) = -0.0002;   R_moc(2,1) = -0.0012;  R_moc(2,2) = 1.0000;

  } catch (EphemerisErr &e) {
    throw IOErr() << "An error occurred while parsing the ephemeris file.\n";
  }
}

/* ----------------------------------------------------------------------
 *               VIZ Site Frame Related Methods
 * ----------------------------------------------------------------------*/
vw::Matrix<double> MOCImageMetadata::GenWorldToLocalTransform(double lat, double lon, double radius)
{
  double sinLat = sin(M_PI * lat / 180.0), cosLat = cos(M_PI * lat / 180.0);
  double xOrigin, yOrigin, zOrigin;
  vw::Vector3 localXAxis, localYAxis, localZAxis;

  // Longitude is defined positive west!
  lon = (360.0 - lon);
  // The following assumes latitude is measured from the equatorial
  // plane with north positive. This is different than normal
  // spherical coordinate conversion where the equivalent angle is
  // measured from the positive z axis.
  xOrigin = radius * cosLat * cos(M_PI * lon / 180.0);
  yOrigin = radius * cosLat * sin(M_PI * lon / 180.0);
  zOrigin = radius * sinLat;

  // Debugging:
  //   cout << "\tImage center: ("
  //        << xOrigin << "," << yOrigin << "," << zOrigin << ")" << endl;

  vw::Matrix<double> translation(4,4);
  translation.set_identity();
  translation[0][2] = xOrigin;
  translation[1][2] = yOrigin;
  translation[2][2] = zOrigin;

  vw::Matrix<double> rotation(4,4);
  rotation.set_identity();

  // Mars local level is a Z-down coordinate system
  localZAxis[0] = -xOrigin / radius;
  localZAxis[1] = -yOrigin / radius;
  localZAxis[2] = -zOrigin / radius;

  if (cosLat != 0.0)
  {
    // Z_local X Z_world  = Y_local
    localYAxis[0] =  localZAxis[1];
    localYAxis[1] = -localZAxis[0];
    localYAxis[2] =  0.0;
    localYAxis /= vw::math::norm_2(localYAxis); // Normalize

    // Y_local X Z_local = X_local
    localXAxis = vw::math::cross_prod(localYAxis, localZAxis);
    localXAxis /= vw::math::norm_2(localXAxis); // Normalize
  }
  else                                     // we're at one of the poles
  {
    if (zOrigin > 0.0)                     // north pole
      localXAxis[0] = -1.0;
    else                                   // south pole
      localXAxis[0] = 1.0;
    localXAxis[1] = 0.0; localXAxis[2] = 0.0;
    localYAxis[0] = 0.0; localYAxis[1] = 1.0; localYAxis[2] = 0.0;
  }

  // The 3x3 matrix consisting of the local level coordinate system
  // axes (in world coordinates) as rows gives the local-to-world
  // rotation:
  rotation[0][0] = localXAxis[0];
  rotation[0][1] = localXAxis[1];
  rotation[0][2] = localXAxis[2];

  rotation[1][0] = localYAxis[0];
  rotation[1][1] = localYAxis[1];
  rotation[1][2] = localYAxis[2];

  rotation[2][0] = localZAxis[0];
  rotation[2][1] = localZAxis[1];
  rotation[2][2] = localZAxis[2];

  // We do the translation followed by rotation:
  vw::Matrix<double> localToWorld = rotation * translation;
  return vw::math::inverse(localToWorld);
}

void MOCImageMetadata::write_viz_site_frame(std::string prefix)
{
  string fileName = prefix + string(".viz");
  std::ofstream vizFile(fileName.c_str());

  vw_out(0) << "MOCImage::WriteVizSiteFrame(): writing viz file..."
            << std::flush;

  vizFile << "#Inventor V2.1 ascii" << std::endl << std::endl;

  vizFile << "DEF +parent1 SoVizGroup" << std::endl;
  vizFile << "{" << std::endl;
  vizFile << "    fields [ SFEnum renderCaching, SFEnum boundingBoxCaching,"
          << std::endl;
  vizFile << "             SFEnum renderCulling, SFEnum pickCulling,"
          << std::endl;
  vizFile << "       SFString title, SFNode parent, SFNode transform,"
    " SFNode render" << std::endl;
  vizFile << "       ]" << std::endl;
  vizFile << "    renderCaching OFF" << std::endl;
  vizFile << "    boundingBoxCaching    OFF" << std::endl;
  vizFile << "    renderCulling ON" << std::endl;
  vizFile << "    pickCulling           ON" << std::endl;
  vizFile << "    title         \"" << m_moc_identifier << "\"" << std::endl;
  vizFile << "    parent                NULL" << std::endl;
  vizFile << "    transform DEF +transform1 MatrixTransform" << std::endl;
  vizFile << "                          {" << std::endl;
  vizFile << "                                  matrix" << std::endl;
  vizFile << "                                  "
          << m_worldToLocal[0][0] << " " << m_worldToLocal[0][1] << " "
          << m_worldToLocal[0][2] << " " << m_worldToLocal[0][3] << std::endl;
  vizFile << "                                  "
          << m_worldToLocal[1][0] << " " << m_worldToLocal[1][1] << " "
          << m_worldToLocal[1][2] << " " << m_worldToLocal[1][3] << std::endl;
  vizFile << "                                  "
          << m_worldToLocal[2][0] << " " << m_worldToLocal[2][1] << " "
          << m_worldToLocal[2][2] << " " << m_worldToLocal[2][3] << std::endl;
  vizFile << "                                  "
          << m_worldToLocal[3][0] << " " << m_worldToLocal[3][1] << " "
          << m_worldToLocal[3][2] << " " << m_worldToLocal[3][3] << std::endl;
  vizFile << "                          }" << std::endl;
  vizFile << "    render DEF +children1 Group" << std::endl;
  vizFile << "    {" << std::endl;
  vizFile << "    }" << std::endl;
  vizFile << "    USE +transform1" << std::endl;
  vizFile << "    USE +children1" << std::endl;
  vizFile << "}" << std::endl;

  vizFile.close();

  vw_out(0) << " done." << std::endl << std::endl;
}

