#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file MOLAReader.cc
///

#include <asp/Sessions/MOC/MOLAReader.h>

#include <iostream>
#include <iomanip>
#include <string>
#include <sstream>
#include <math.h>

#include "boost/filesystem/operations.hpp"
#include "boost/filesystem/fstream.hpp"
namespace fs = boost::filesystem;

using namespace std;
using namespace vw;

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*        General Utilities for parsing data from a PEDR         */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

// Takes a four byte MOLA PEDR data record, with MSB first, and
// returns a reconstructed unsigned integer.
static inline unsigned int uchar_to_uint(const unsigned char* record) {
  return ((unsigned int)record[0] << 24) + ((unsigned int)record[1] << 16) +
         ((unsigned int)record[2] << 8)  + ((unsigned int)record[3]);
}


// Takes a four byte MOLA PEDR data record, with MSB first, and
// returns a reconstructed SIGNED integer.
static inline int uchar_to_int(const unsigned char* record) {
  return ((int)record[0] << 24) + ((int)record[1] << 16) +
          ((int)record[2] << 8) + ((int)record[3]);
}

inline double uchars_to_ephemeris_time( const unsigned char* secs,
                                        const unsigned char* msecs ) {
  double seconds = (double) uchar_to_int(secs);
  double microsecs = (double) uchar_to_int(msecs);

  //  printf(" -->  %lf  %lf\n", seconds, microsecs);
  return seconds + (microsecs * 1.0e-6);
}

inline float uchar_to_latlon ( const unsigned char* record ) {
  return ((float) uchar_to_int(record)) / 1.0e6;
}

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                    PEDR_Shot Class Methods                    */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

#define FRAME_OFFSET 9.5

PEDR_Shot::PEDR_Shot() : PEDR_Frame() { shot_id = 0; }

PEDR_Shot::PEDR_Shot(unsigned char* pedr_record) : PEDR_Frame(pedr_record) {
  throw MOLA_PEDR_Err() << "Cannot create a MOLA shot without specifying the shot number.";
}

PEDR_Shot::PEDR_Shot(unsigned char* pedr_record,
                     unsigned short shot_number) : PEDR_Frame(pedr_record) {
  if (shot_number > 19) {
    throw MOLA_PEDR_Err() << "Invalid MOLA shot number.  There are only 20 shots per MOLA frame.";
  }
  shot_id = shot_number;
}

// Corrected ephemeris time for this MOLA shot
double PEDR_Shot::ephemeris_time() {
  return PEDR_Frame::ephemeris_time() + ((double)shot_id - FRAME_OFFSET) * shot_dt();
}

// Corrected latitude for this MOLA shot
float PEDR_Shot::areo_latitude() {
  float d_lat = uchar_to_latlon( &pedr_datum[DELTA_LATITUDE] );
  return PEDR_Frame::areo_latitude() + ((float) shot_id - FRAME_OFFSET) / 20.0 * d_lat;
}

// Corrected longitude for this MOLA shot
float PEDR_Shot::areo_longitude() {
  float d_lon = uchar_to_latlon( &pedr_datum[DELTA_LONGITUDE] );
  return PEDR_Frame::areo_longitude() + ((float) shot_id - FRAME_OFFSET) / 20.0 * d_lon;
}

// Planetary radius mesaured for this shot
float PEDR_Shot::shot_planetary_radius() {
  unsigned char* frame_radii = &pedr_datum[SHOT_PLANETARY_RADIUS];
  return (float) uchar_to_uint(&frame_radii[4*shot_id]) / 100.0;  // 4-byte record
}

// Corrected oeuoeuf the Goddard Mars Model 3 for this frame
float PEDR_Shot::areoid_radius() {
  return PEDR_Frame::areoid_radius() +
    ((float) shot_id - FRAME_OFFSET) / 20.0 * PEDR_Frame::delta_areoid();
}

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                    PEDR_Frame Class Methods                    */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
PEDR_Frame::PEDR_Frame() {
  // Fill the PEDR record with NULL data
  for (unsigned int i = 0; i < PEDR_RECORD_SIZE; i++) {
    pedr_datum[i] = 0;
  }
}

PEDR_Frame::PEDR_Frame(unsigned char* record) {
  // Make a local copy of the PEDR record
  for (unsigned int i = 0; i < PEDR_RECORD_SIZE; i++) {
    pedr_datum[i] = record[i];
  }
}

// Use this function to get a list of 20 MOLA shots for this
// PEDR Frame.
std::list<class PEDR_Shot> PEDR_Frame::MOLA_shots() {

  unsigned short i;
  std::list<PEDR_Shot> shot_list;

  for (i = 0; i < PEDR_SHOTS_PER_FRAME; i++) {
    PEDR_Shot s(pedr_datum, i);  /* Create a MOLA_Shot */
    shot_list.push_back(s);
  }

  return shot_list;
}

double PEDR_Frame::ephemeris_time() {
  return uchars_to_ephemeris_time(&pedr_datum[EPHEMERIS_SECS],
                                  &pedr_datum[EPHEMERIS_MSECS]);
}

double PEDR_Frame::start_time() {
  return ephemeris_time() - FRAME_OFFSET * shot_dt();
}

double PEDR_Frame::end_time() {
  return ephemeris_time() + FRAME_OFFSET * shot_dt();
}

double PEDR_Frame::shot_dt() {
  return 0.1;  // NOMINAL MOLA SHOT RATE IS 10HZ
}

unsigned int PEDR_Frame::orbit_reference_nmuber() {
  return uchar_to_uint(&pedr_datum[ORBIT_NUMBER]);
}

float PEDR_Frame::areo_latitude() {
  return uchar_to_latlon(&pedr_datum[AREO_LAT]);
}

float PEDR_Frame::areo_longitude() {
  return uchar_to_latlon(&pedr_datum[AREO_LON]);
}

// Distance from Mars center of mass to MGS
float PEDR_Frame::mgs_radial_distance() {
  return (float) uchar_to_uint(&pedr_datum[RADIAL_DIST]) / 100.0;
}

// Average of the planetary radii measured for these 20 shots
float PEDR_Frame::shot_planetary_radius() {
  return (float) uchar_to_uint(&pedr_datum[MDPT_PLANET_RADIUS]) / 100.0;
}

/// Radius of the Goddard Mars Model 3 for this frame
float PEDR_Frame::areoid_radius() {
  return (float) uchar_to_uint(&pedr_datum[AREOID_RADIUS]) / 100.0;
}

// Average change in the areoid height across the 20 MOLA shots in this frame
float PEDR_Frame::delta_areoid() {
  return (float) uchar_to_int(&pedr_datum[DELTA_AREOID]) / 100.0;
}

// Average change in the latitude across the 20 MOLA shots in this frame
float PEDR_Frame::delta_latitude() {
  return (float) uchar_to_latlon(&pedr_datum[DELTA_LATITUDE]);
}

// Average change in the longitude across the 20 MOLA shots in this frame
float PEDR_Frame::delta_longitude() {
  return (float) uchar_to_latlon(&pedr_datum[DELTA_LONGITUDE]);
}

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                  MOLA_PEDR_Reader Class Methods               */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */


MOLA_PEDR_Reader::MOLA_PEDR_Reader( std::string const& pedr_data_directory,
                                    unsigned int orbit_number) {

  ostringstream filename;
  unsigned int n = 0;

  filename << pedr_data_directory << "/ap"
           <<(orbit_number + 10000) / 1000 << "xxx/ap"
           << orbit_number + 10000 << "l.b";

  // Search for the PEDR file that has the orbit number that is
  // less than, but closest to, the orbit we are looking for.
  pedr_file = NULL;
  //  std::cout << "MOLA attempting to open " << filename.str() << std::endl;
  while (!(  pedr_file = fopen(filename.str().c_str(), "r"))) {
    long end = filename.tellp();
    filename.seekp(end - 16);
    filename << (orbit_number + 10000 - n) / 1000 << "xxx/ap"
             <<  orbit_number + 10000 - n << "l.b";
    n++;

    //    std::cout << "MOLA attempting to open " << filename.str() << "   " << n << "    " << orbit_number << std::endl;

    // If we have tried all of the orbit numbers in the directory,
    // just give up.
    if (n >= orbit_number) {
      throw MOLA_PEDR_Err() <<  "Could not find MOLA database file.";
    }
  }
  // Read ascii header.
  fread (pedr_header, PEDR_HEADER_SIZE, 1, pedr_file);
  cout << "\tSuccesfully opened MOLA PEDR record: " << filename.str() << ".\n";
}

MOLA_PEDR_Reader::~MOLA_PEDR_Reader() {
  if (pedr_file) {
    fclose(pedr_file);
  }
}

/* Reset the file reader to the beginning of the file */
void MOLA_PEDR_Reader::reset_filepointer() {
  fseek(pedr_file, PEDR_HEADER_SIZE, 0);
}


/*
 * Search for a record that is within tol seconds of the
 * target time.  The time reference is J2000 epoch (i.e.
 * "ephemeris time."
 */
PEDR_Shot MOLA_PEDR_Reader::get_pedr_by_time(double target_time,
                                             double tolerance) {

  double closest_et = 0.0;
  list<PEDR_Shot> nearby_shot_list;
  list<PEDR_Shot>::iterator shot_iterator;
  PEDR_Shot closest_shot;
  nearby_shot_list = get_pedr_by_time_range(target_time - tolerance / 2.0,
                                            target_time + tolerance / 2.0);

  // If there were no matches, throw an exception
  if (nearby_shot_list.size() == 0)
    throw MOLA_PEDR_Err() << "Could not find a shot that matched the target time.";

  // Otherwise, look for the closest match
  for(shot_iterator = nearby_shot_list.begin();
      shot_iterator != nearby_shot_list.end();
      shot_iterator++) {

    if (fabs((*shot_iterator).ephemeris_time() - target_time) <
        fabs(closest_et - target_time)) {
      closest_et = (*shot_iterator).ephemeris_time();
      closest_shot = *shot_iterator;
    }
  }

  return closest_shot;
}

std::list<PEDR_Shot>
MOLA_PEDR_Reader::get_pedr_by_time_range(double et_start, double et_end) {

  unsigned char current_pedr[PEDR_RECORD_SIZE];
  list<PEDR_Shot> current_shot_list;
  list<PEDR_Shot> selected_shot_list;
  list<PEDR_Shot>::iterator shot_iterator;

  /* Seek to the beginning of the data records */
  reset_filepointer();

  /* Search through the file and find the best match. */
  while (fread((char*) current_pedr, PEDR_RECORD_SIZE, 1, pedr_file)) {

    /* Extract the timestamp for the current ephemeris entry */
    PEDR_Frame frame(current_pedr);

    /*
     * First off, check to make sure that we haven't gone
     * past a time index in the file that is greater than
     * et_end.  If we have, we can safely terminate the file
     * search.
     */
    if (frame.start_time() > et_end)
      break;

    /*
     * Optimization: don't bother parsing frames that
     * do not contain the time ranges of interest.
     */
    if (frame.end_time() < et_start)
      continue;

    /*
     * We have to parse this PEDR frame into individual
     * shots and check each one individually to see if it falls
     * in the specified time range.  If it does, add it to the
     * list of selected shots.
     */
    current_shot_list = frame.MOLA_shots();
    for(shot_iterator = current_shot_list.begin();
        shot_iterator != current_shot_list.end();
        shot_iterator++) {

      if ((*shot_iterator).ephemeris_time() > et_start &&
          (*shot_iterator).ephemeris_time() < et_end) {
        selected_shot_list.push_back(*shot_iterator);
      }

    }
  }

  /* If there were no matches, throw an exception */
  if (selected_shot_list.size() == 0)
    throw MOLA_PEDR_Err() << "Could not find a shot that matched the target time range.";
  return selected_shot_list;
}


/*
 * Search for PEDR entries that lie within a region bounded by
 * the box given below.
 *
 * IMPORTANT NOTE:  Coordinates must be given in a planetocentric,
 * east positive frame of reference so that they are compatible with
 * the MOLA standard.
 *
 * Unfortunately this search method is not very fast (it takes several
 * seconds per orbit).
 */
std::list<PEDR_Shot>
MOLA_PEDR_Reader::get_pedr_by_areo_latlon(float north, float south,
                                          float east, float west) {


  float current_lat, current_lon;
  unsigned char current_pedr[PEDR_RECORD_SIZE];
  list<PEDR_Shot> selected_shot_list;
  list<PEDR_Shot> current_shot_list;
  list<PEDR_Shot>::iterator shot_iterator;

  /* Seek to the beginning of the data records */
  reset_filepointer();

  /* Search through the file and find the best match. */
  while (fread((char*) current_pedr, PEDR_RECORD_SIZE, 1, pedr_file)) {

    /* Extract the timestamp for the current ephemeris entry */
    PEDR_Frame frame(current_pedr);

    /*
     * We have to parse this PEDR frame into individual
     * shots and check each one individually to see if it falls
     * in the specified lat/lon range.  If it does, add it to the
     * list of selected shots.
     */
    current_shot_list = frame.MOLA_shots();
    for(shot_iterator = current_shot_list.begin();
        shot_iterator != current_shot_list.end();
        shot_iterator++) {

      current_lat = (*shot_iterator).areo_latitude();
      current_lon = (*shot_iterator).areo_longitude();

      if ((current_lat > south) && (current_lat < north) &&
          (current_lon > west) && (current_lon < east)) {
        selected_shot_list.push_back((*shot_iterator));
      }
    }
  }

  return selected_shot_list;
}


