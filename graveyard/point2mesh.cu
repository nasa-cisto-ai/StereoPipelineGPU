#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file point2mesh.cc
///

#ifdef _MSC_VER
#pragma warning(disable:4244)
#pragma warning(disable:4267)
#pragma warning(disable:4996)
#endif

#ifdef NDEBUG
#undef NDEBUG
#endif

#include <stdlib.h>

#include <boost/program_options.hpp>
namespace po = boost::program_options;

#include <vw/FileIO.h>
#include <vw/Image.h>
#include <vw/Math.h>
using namespace vw;

#include <asp/Tools/nff_terrain.h>

// Erases a file suffix if one exists and returns the base string
static std::string prefix_from_pointcloud_filename(std::string const& filename) {
  std::string result = filename;

  // First case: filenames that match <prefix>-PC.<suffix>
  int index = result.rfind("-PC.");
  if (index != -1) {
    result.erase(index, result.size());
    return result;
  }

  // Second case: filenames that match <prefix>.<suffix>
  index = result.rfind(".");
  if (index != -1) {
    result.erase(index, result.size());
    return result;
  }

  // No match
  return result;
}

/// Erases a file suffix if one exists and returns the base string
static std::string prefix_from_filename(std::string const& filename) {
  std::string result = filename;
  int index = result.rfind(".");
  if (index != -1)
    result.erase(index, result.size());
  return result;
}

/// Erases a file suffix if one exists and returns the base string
static std::string suffix_from_filename(std::string const& filename) {
  std::string result = filename;
  int index = result.rfind(".");
  if (index != -1)
    result.erase(0, index);
  return result;
}

// Allows FileIO to correctly read/write these pixel types
namespace vw {
  template<> struct PixelFormatID<Vector3>   { static const PixelFormatEnum value = VW_PIXEL_GENERIC_3_CHANNEL; };
}


template <class ViewT>
BBox<3> pointcloud_bbox(ImageViewBase<ViewT> const& point_image) {
  // Compute bounding box
  BBox<3> result;
  typename ViewT::pixel_accessor row_acc = point_image.impl().origin();
  for( int32 row=0; row < point_image.impl().rows(); ++row ) {
    typename ViewT::pixel_accessor col_acc = row_acc;
    for( int32 col=0; col < point_image.impl().cols(); ++col ) {
      if (*col_acc != Vector3())
        result.grow(*col_acc);
      col_acc.next_col();
    }
    row_acc.next_row();
  }
  return result;
}



// Apply an offset to the points in the PointImage
class PointOffsetFunc : public UnaryReturnSameType {
  Vector3 m_offset;

public:
  PointOffsetFunc(Vector3 const& offset) : m_offset(offset) {}

  template <class T>
  T operator()(T const& p) const {
    if (p == T()) return p;
    return p + m_offset;
  }
};

template <class ImageT>
UnaryPerPixelView<ImageT, PointOffsetFunc>
inline point_image_offset( ImageViewBase<ImageT> const& image, Vector3 const& offset) {
  return UnaryPerPixelView<ImageT,PointOffsetFunc>( image.impl(), PointOffsetFunc(offset) );
}



class PointTransFunc : public ReturnFixedType<Vector3> {
  Matrix3x3 m_trans;
public:
  PointTransFunc(Matrix3x3 trans) : m_trans(trans) {}
  Vector3 operator() (Vector3 const& pt) const { return m_trans*pt; }
};

int main( int argc, char *argv[] ) {
  set_debug_level(VerboseDebugMessage+11);

  std::string pointcloud_filename, out_prefix = "", output_file_type, texture_filename;
  unsigned cache_size, max_triangles;
  float mesh_tolerance;
  unsigned simplemesh_h_step, simplemesh_v_step;
  int debug_level;
  double phi_rot, omega_rot, kappa_rot;
  std::string rot_order;

  po::options_description desc("Options");
  desc.add_options()
    ("help", "Display this help message")
    ("simple-mesh", "Generate simple (non-adaptive) mesh")
    ("simplemesh-h-step", po::value<unsigned>(&simplemesh_h_step)->default_value(1), "Horizontal step size for simple meshing algorithm")
    ("simplemesh-v-step", po::value<unsigned>(&simplemesh_v_step)->default_value(1), "Vertical step size for simple meshing algorithm")
    ("mesh-tolerance", po::value<float>(&mesh_tolerance)->default_value(0.001), "Tolerance for the adaptive meshing algorithm")
    ("max_triangles", po::value<unsigned>(&max_triangles)->default_value(1000000), "Maximum triangles for the adaptive meshing algorithm")
    ("center", "Center the model around the origin.  Use this option if you are experiencing numerical precision issues.")
    ("cache", po::value<unsigned>(&cache_size)->default_value(2048), "Cache size, in megabytes")
    ("input-file", po::value<std::string>(&pointcloud_filename), "Explicitly specify the input file")
    ("texture-file", po::value<std::string>(&texture_filename), "Specify texture filename")
    ("grayscale-texture", "Use grayscale image processing when modifying the texture image (for .iv and .vrml files only)")
    ("output-prefix,o", po::value<std::string>(&out_prefix), "Specify the output prefix")
    ("output-filetype,t", po::value<std::string>(&output_file_type)->default_value("ive"), "Specify the output file")
    ("debug-level,d", po::value<int>(&debug_level)->default_value(vw::DebugMessage-1), "Set the debugging output level. (0-50+)")

    ("rotation-order", po::value<std::string>(&rot_order)->default_value("xyz"),"Set the order of an euler angle rotation applied to the 3D points prior to DEM rasterization")
    ("phi-rotation", po::value<double>(&phi_rot)->default_value(0),"Set a rotation angle phi")
    ("omega-rotation", po::value<double>(&omega_rot)->default_value(0),"Set a rotation angle omega")
    ("kappa-rotation", po::value<double>(&kappa_rot)->default_value(0),"Set a rotation angle kappa")
                ("flip-triangles", "Use clockwise vertex ordering (.iv and .tri files only)");

  po::positional_options_description p;
  p.add("input-file", 1);
  p.add("texture-file", 1);

  po::variables_map vm;
  po::store( po::command_line_parser( argc, argv ).options(desc).positional(p).run(), vm );
  po::notify( vm );

  // Set the Vision Workbench debug level
  //set_debug_level(debug_level);
  //vw_system_cache().resize( cache_size*1024*1024 );

  std::ostringstream usage;
  usage << "Usage: " << argv[0] << " [options] <pointcloud> <texture file> ...";
  usage << std::endl << std::endl << desc << std::endl;

  if( vm.count("help") ) {
    vw_out(0) << usage.str();
    return 1;
  }

  if( vm.count("input-file") != 1 || (vm.count("texture-file") != 1 && output_file_type != "tri") ) {
    vw_out(0) << "Error: Must specify exactly one pointcloud file and one texture file!" << std::endl;
    vw_out(0) << usage.str();
    return 1;
  }

  if( out_prefix == "" ) {
    out_prefix = prefix_from_pointcloud_filename(pointcloud_filename);
  }

  DiskImageView<Vector3> point_disk_image(pointcloud_filename);
  ImageViewRef<Vector3> point_image = point_disk_image;

  if (vm.count("center")) {
    BBox<float,3> bbox = pointcloud_bbox(point_disk_image);
    std::cout << "\t--> Centering model around the origin.\n";
    std::cout << "\t    Initial point image bounding box: " << bbox << "\n";
    Vector3 midpoint = (bbox.max() + bbox.min()) / 2.0;
    std::cout << "\t    Midpoint: " << midpoint << "\n";
    point_image = point_image_offset(point_image, -midpoint);
    BBox<float,3> bbox2 = pointcloud_bbox(point_image);
    std::cout << "\t    Re-centered point image bounding box: " << bbox2 << "\n";
  }

  // Apply an (optional) rotation to the 3D points before building the mesh.
  if (phi_rot != 0 || omega_rot != 0 || kappa_rot != 0) {
    std::cout << "Applying rotation sequence: " << rot_order << "      Angles: " << phi_rot << "   " << omega_rot << "  " << kappa_rot << "\n";
    Matrix3x3 rotation_trans = math::euler_to_rotation_matrix(phi_rot,omega_rot,kappa_rot,rot_order);
    point_image = per_pixel_filter(point_image, PointTransFunc(rotation_trans));
  }

  std::cout << "\nGenerating 3D mesh from point cloud:\n";
  Mesh mesh_maker;
  if(vm.count("simple-mesh")) {
    mesh_maker.build_simple_mesh(point_image, simplemesh_h_step, simplemesh_v_step);
  } else {
    mesh_maker.build_adaptive_mesh(point_image, mesh_tolerance, max_triangles);
  }

  // New style (open scene graph) 3D models
  if (output_file_type == "ive")
    mesh_maker.write_osg(out_prefix+".ive", texture_filename);

  // Old style (open inventor) 3D models
  else if(output_file_type == "iv" || output_file_type == "vrml") {
    std::string corrected_texture_filename = prefix_from_filename(texture_filename)+".jpg";
    if (suffix_from_filename(texture_filename) != "jpg")
      if (vm.count("grayscale-texture") ) {
        DiskImageView<PixelGray<uint8> > texture(texture_filename);
        write_image(corrected_texture_filename, texture);
      } else {
        DiskImageView<PixelRGB<uint8> > texture(texture_filename);
        write_image(corrected_texture_filename, texture);
      }

    if(output_file_type == "iv")
      mesh_maker.write_inventor(out_prefix+".iv", corrected_texture_filename, vm.count("flip-triangles") != 0);

    if(output_file_type == "vrml")
      mesh_maker.write_vrml(out_prefix+".vrml", corrected_texture_filename);

  }

  // Simple trimesh (Stanford robotics libraries) 3D models
  else if (output_file_type == "tri") {
    mesh_maker.write_trimesh(out_prefix+".tri", vm.count("flip-triangles") != 0);
  } else {
    std::cout << "Unsupported 3D file type.\n";
  }

  return 0;
}
