#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file ctximage.cc
///

#ifdef _MSC_VER
#pragma warning(disable:4244)
#pragma warning(disable:4267)
#pragma warning(disable:4996)
#endif

#ifdef NDEBUG
#undef NDEBUG
#endif

#include <stdlib.h>

#include <boost/program_options.hpp>
namespace po = boost::program_options;

#include <vw/vw.h>
using namespace vw;
using namespace vw::cartography;

#include <asp/Core/DiskImageResourceDDD.h>

/// Erases a file suffix if one exists and returns the base string
static std::string prefix_from_filename(std::string const& filename) {
  std::string result = filename;
  int index = result.rfind(".");
  if (index != -1)
    result.erase(index, result.size());
  return result;
}

//  mask_zero_pixels()
//
struct MaskZeroPixelFunc: public vw::UnaryReturnSameType {

  template <class PixelT>
  PixelT operator() (PixelT const& pix) const {
    if (pix.v() == 0)
      return PixelT();  // Mask pixel
    else
      return PixelT(pix.v());
  }
};

template <class ViewT>
vw::UnaryPerPixelView<ViewT, MaskZeroPixelFunc>
mask_zero_pixels(vw::ImageViewBase<ViewT> const& view) {
  return vw::per_pixel_filter(view.impl(), MaskZeroPixelFunc());
}

// *******************************************************************
// normalize()
// *******************************************************************

/// \cond INTERNAL
template <class PixelT>
class RescalePixelsWithAlphaFunc: public UnaryReturnSameType {
  typedef typename CompoundChannelType<PixelT>::type channel_type;
  channel_type m_old_min, m_new_min,m_old_max, m_new_max;
  double m_old_to_new_ratio;
public:
  RescalePixelsWithAlphaFunc( channel_type old_min, channel_type old_max,
                              channel_type new_min, channel_type new_max )
    : m_old_min(old_min), m_new_min(new_min), m_old_max(old_max), m_new_max(new_max)
  {
    if( old_max == old_min ) { m_old_to_new_ratio = 0.0; }
    else { m_old_to_new_ratio = (new_max - new_min)/(double)(old_max - old_min); }
  }

  PixelT operator()( PixelT value ) const {
    PixelT result;
    for (int i = 0; i < CompoundNumChannels<PixelT>::value-1; ++i) {
      result[i] = ((value[i] - m_old_min) * m_old_to_new_ratio + m_new_min);
      // Limit to the min and max values
      if (result[i] > m_new_max) result[i] = m_new_max;
      if (result[i] < m_new_min) result[i] = m_new_min;
    }
    // Copy the alpha value
    result[CompoundNumChannels<PixelT>::value-1] = value[CompoundNumChannels<PixelT>::value-1];
    return result;
  }
};
/// \endcond

/// Renormalize the values in an image to fall within the range [low,high).
template <class ImageT, class ValT>
UnaryPerPixelView<ImageT, RescalePixelsWithAlphaFunc<typename ImageT::pixel_type> >
inline rescale_pixels_with_alpha( ImageViewBase<ImageT> const& image, ValT old_low, ValT old_high, ValT new_low, ValT new_high ) {
  typedef RescalePixelsWithAlphaFunc<typename ImageT::pixel_type> func_type;
  func_type func(old_low, old_high, new_low, new_high );
  return UnaryPerPixelView<ImageT, func_type >( image.impl(), func );
}


int main( int argc, char *argv[] ) {

  // The DiskImageResourceDDD needs to be registered since it is not a
  // built-in VW FileIO driver.
  vw::DiskImageResource::register_file_type( ".ddd", vw::DiskImageResourceDDD::type_static(), &vw::DiskImageResourceDDD::construct_open, &vw::DiskImageResourceDDD::construct_create );


  std::string input_file_name, output_file_name, index_file_name;
  int debug_level;

  po::options_description desc("Options");
  desc.add_options()
    ("help", "Display this help message")
    ("input-file", po::value<std::string>(&input_file_name), "Explicitly specify the input file")
    ("index-file", po::value<std::string>(&index_file_name)->default_value("none"), "Specify the index file")
    ("output-file,o", po::value<std::string>(&output_file_name)->default_value("none"), "Specify the output file")
    ("debug-level", po::value<int>(&debug_level)->default_value(InfoMessage), "Set the level of debugging output.");
  po::positional_options_description p;
  p.add("input-file", 1);
  p.add("index-file", 2);

  po::variables_map vm;
  po::store( po::command_line_parser( argc, argv ).options(desc).positional(p).run(), vm );
  po::notify( vm );

  if( vm.count("help") ) {
    std::cout << "Usage: " << argv[0] << " <ddd file> <catalog file>\n";
    std::cout << desc << "\n";
    return 1;
  }

  if( vm.count("input-file") != 1 ) {
    std::cout << "Error: Must specify exactly one input file!\n\n";
    std::cout << "Usage: " << argv[0] << " <ddd file> <catalog file>\n";
    std::cout << desc << "\n";
    return 1;
  }

  if( vm.count("index-file") != 1 ) {
    std::cout << "Error: Must specify exactly one index file!\n\n";
    std::cout << "Usage: " << argv[0] << " <ddd file> <catalog file>\n";
    std::cout << desc << "\n";
    return 1;
  }

  if (output_file_name == "none") {
    output_file_name = prefix_from_filename(input_file_name) + ".tif";
  }

  // Set the vision workbench debugging output level.
  set_debug_level(debug_level);

  // The following big 'ol chunk of code fetches the sun incidence
  // angle from the ctx catalog for use in photometric calibration.
  double emission, incidence, phase;
  bool m_do_photometric_calibartion = false;
  if( index_file_name != "none" ) {
    std::cout << "Using index file: " << index_file_name << "\n";
    std::string prefix = prefix_from_filename(input_file_name);

    std::ifstream input(index_file_name.c_str());
    if (!(input.good())) {
      std::cout << "Could not open index file: " << index_file_name << "\nExiting.\n\n";
      exit(1);
    }

    char c_line[2048];
    bool found = false;
    while (!input.eof()) {
      input.getline(c_line, 2048);
      std::string line = c_line;
      if (line.find(prefix) == 0) {
        // Split into several strings using whitespace as delimeter
        std::vector<std::string> split_vec, final_vec;
        boost::split(split_vec, line, boost::is_space());
        // Get rid of empty strings from multiple adjacent whitespaces
        for (unsigned int i=0; i < split_vec.size();++i) {
          if (split_vec[i].size() != 0)
            final_vec.push_back(split_vec[i]);
        }
        if (final_vec.size() != 6) {
          std::cout << "Error parsing line: " << line << "\nExiting.\n\n";
          exit(1);
        }
        emission = atof(final_vec[3].c_str());
        incidence = atof(final_vec[4].c_str());
        phase = atof(final_vec[5].c_str());
        std::cout << "Photometric Values -- Emission: " << emission << "  Incidence: " << incidence << "  Phase: " << phase << "\n";
        found = true;
        break;
      }
    }
    if (!found) {
      std::cout << "Could not find index entry for image: " << prefix << "\nExiting.\n\n";
      exit(1);
    }
    input.close();

    m_do_photometric_calibartion = true;
  }

  try {
    ImageView<PixelGrayA<uint16> > disk_image;
    read_image(disk_image, input_file_name);

    // Use the grassfire algorithm to force the black border
    // information to be transparent.
    select_channel(disk_image,1) = clamp(grassfire(select_channel(disk_image,0)),0,1) * 65535;

    // Fetch parameters from the CTX header
    DiskImageResourceDDD file_resource( input_file_name );
    double fullwidth = atol(file_resource.query("projection_fullwidth").c_str());
    double projection_x_offset = atol(file_resource.query("projection_x_offset").c_str());
    double projection_y_offset = atol(file_resource.query("projection_y_offset").c_str());
    double exposure = atof(file_resource.query("exposure").c_str());

    // Set up a georefernce object so that we can embed this
    // information in the file.  Note that we choose a "dummy"
    // mercator projection here simply so that we put something in the
    // geotiff header.  The choice of projection doesn't matter in the
    // end because we pass these images along to geoblend, which only
    // uses the affine transform information anyway.
    GeoReference georef;
    georef.set_mercator(0,0,1); // dummy projection
    Matrix3x3 geotransform = math::identity_matrix<3>();
    geotransform(0,2) = projection_x_offset;
    geotransform(1,2) = projection_y_offset;
    georef.set_transform(geotransform);

    if (m_do_photometric_calibartion) {
      float sun_coeff = cos(incidence*M_PI/180);

      // These normalization factors were chosen based on sampling
      // some representative light and dark images.  They produced
      // reasonable results for this particular mosaic, but there is
      // probably a more principled way to choose these.
      float norm_low = 5.3;
      float norm_high = 7.8;

      if (sun_coeff < 0.1) sun_coeff = 0.1; // Prevent division by a
                                            // number close to zero
                                            // for very larg incidence
                                            // angles
      ImageViewRef<PixelGrayA<uint8> > corrected_image = channel_cast<uint8>(normalize(clamp(log(1+(disk_image / (exposure*sun_coeff))),norm_low,norm_high),norm_low,norm_high,0.0,255.0));
      DiskImageResourceGDAL r = DiskImageResourceGDAL( output_file_name, corrected_image.format(), Vector2i(vw_settings().default_tile_size(),vw_settings().default_tile_size()) );
      write_georeference( r, georef );
      write_image(r, corrected_image, TerminalProgressCallback() );
    } else {

      // See comment above about choosing these normalization factors.
      float norm_low = 6;
      float norm_high = 8;

      ImageViewRef<PixelGrayA<uint8> > corrected_image = channel_cast<uint8>(normalize(clamp(log(disk_image/exposure),norm_low,norm_high),norm_low,norm_high,0.0,255.0));
      DiskImageResourceGDAL r = DiskImageResourceGDAL( output_file_name, corrected_image.format(), Vector2i(vw_settings().default_tile_size(),vw_settings().default_tile_size()) );
      write_georeference( r, georef );
      write_image(r, corrected_image, TerminalProgressCallback() );
    }
  }
  catch( Exception& e ) {
    std::cerr << "Error: " << e.what() << std::endl;
  }

  return 0;
}
