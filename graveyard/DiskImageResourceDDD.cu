#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file DiskImageResourceDDD.cc
///
#ifdef _MSC_VER
#pragma warning(disable:4244)
#pragma warning(disable:4267)
#pragma warning(disable:4996)
#endif

#include <vector>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>

#ifdef __APPLE__
static const int BITSPERBYTE = 8;
#else
#include <values.h>                        // for BITSPERBYTE
#endif

#include <boost/algorithm/string.hpp>

#include <vw/Core/Exception.h>
#include <asp/Core/DiskImageResourceDDD.h>

using namespace std;
using namespace boost;

namespace vw
{
  static const int IMAGE_HEADER_LENGTH = 1024;
  static const int IMAGE_LABEL_OFFSET = 24;
  static const int IMAGE_LABEL_LENGTH = IMAGE_HEADER_LENGTH-IMAGE_LABEL_OFFSET;
  static const uint32 MAGIC = 1659;

  struct DDDHeader
  {
    uint32 magic;                                  // always set to MAGIC
    uint32 numScanlines, bytesPerScanline;
    uint32 bitsPerElement; // the data size unless there's padding
    uint32 spare1, spare2;
    char label[IMAGE_LABEL_LENGTH];
  };

  // - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
  // The label contains linefeed delimited strings, e.g.:
  //
  // decompressed-from ./4A_04_1001004F00_01.DAT
  // id 79 time 844381966:250
  // start 0 cross 5056 down 7168
  // cam ctx
  // mode 0x0
  // dac 195
  // offset 234 232
  // sram_base 0
  // start_addr 0
  // exposure 1.87 msec (19)
  // FPA-temp 17.9 C (2395)
  // - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -


  std::string
  DiskImageResourceDDD::query(std::string const& key) const
  {
    std::map<std::string, std::string>::const_iterator
      entry = m_header_entries.find(key);

    if (entry != m_header_entries.end())
      return (*entry).second;
    else
      throw NotFoundErr() << "DiskImageResourceDDD::query(): "
                          << "no matching value found for \"" << key << "\"";
  }

  void
  DiskImageResourceDDD::parse_ddd_header(const char* header_label)
  {
    string header_label_string(header_label);
    istringstream label_stream(header_label_string);

    while (!label_stream.eof())
    {
      char line[IMAGE_LABEL_LENGTH+1];
      vector<string> tokens;

      label_stream.getline(line, sizeof(line));
      split(tokens, line, is_any_of(" "));

      // First handle any special cases... yes this is a bit hacky,
      // but their label format is a bit hacky...
      if (tokens[0] == string("id"))
      {
        m_header_entries[tokens[2]] = tokens[3];
      }
      else if (tokens[0] == string("start"))
      {
        m_header_entries[tokens[2]] = tokens[3];
        m_header_entries[tokens[4]] = tokens[5];
      }
      else if (tokens[0] == string("offset"))
      {
        tokens[1] = tokens[1] + tokens[2];
      }
      else if ((tokens[0] == string("exposure")) ||
               (tokens[0] == string("FPA-temp")))
      {
        tokens[1] = tokens[1] + tokens[2] + tokens[3];
      }
      else if (tokens[0] == string("@stereographic"))
      {
        std::cout << "Projection line: " << line << "\n";
        m_header_entries["projection"] = "stereographic";
        m_header_entries["projection_x_offset"] = tokens[1];
        m_header_entries["projection_y_offset"] = tokens[2];
        m_header_entries["projection_fullwidth"] = tokens[3];
        m_header_entries["projection_pole_flag"] = tokens[4];
        m_header_entries["projection_extent"] = tokens[5];
      }

      m_header_entries[tokens[0]] = tokens[1];
    }
  }

  inline uint32
  swap_long(uint32 data)
  {
    unsigned char *bytes = reinterpret_cast<unsigned char *>(&data);
    unsigned char bytes2 = bytes[2];
    unsigned char bytes3 = bytes[3];

    bytes[3] = bytes[0];
    bytes[2] = bytes[1];
    bytes[1] = bytes2;
    bytes[0] = bytes3;

    return data;
  }

  inline void
  swap_longs(void *buffer, size_t numLongs)
  {
    uint32 *longs = reinterpret_cast<uint32 *>(buffer);
    uint32 *endLong = longs + numLongs;

    while (longs != endLong)
    {
      uint8 *bytes = (uint8 *)(longs);
      uint8 bytes2 = bytes[2];
      uint8 bytes3 = bytes[3];

      bytes[3] = bytes[0];
      bytes[2] = bytes[1];
      bytes[1] = bytes2;
      bytes[0] = bytes3;

      longs++;
    }
  }

  inline void
  swap_shorts(void *buffer, size_t numShorts)
  {
    uint16 *shorts = reinterpret_cast<uint16 *>(buffer);
    uint16 *endShort = shorts + numShorts;

    while (shorts != endShort)
    {
      uint8 *bytes = (uint8 *)(shorts);
      uint8 bytes1 = bytes[1];
      bytes[1] = bytes[0];
      bytes[0] = bytes1;
      shorts++;
    }
  }

  // Set the default block size to be the width of the image by 10
  // scanlines.
  Vector2i DiskImageResourceDDD::block_size() const
  {
    return Vector2i(m_format.cols,10);
  }

  /// Bind the resource to a file for writing.
  void
  DiskImageResourceDDD::create(std::string const& filename,
                               ImageFormat const& format)
  {
    throw NoImplErr()
      << "The DDD driver does not yet support creation of DDD files";
  }

  /// Bind the resource to a file for reading.  Confirm that we can open
  /// the file and that it has a sane pixel format.
  void
  DiskImageResourceDDD::open(std::string const& filename)
  {
    m_filename = filename;
    ifstream image_file(m_filename.c_str(), ios::in | ios::binary);

    DDDHeader header;
    image_file.read((char *)(&header), sizeof(DDDHeader));

    if (image_file.bad())
      throw IOErr() << "DiskImageResourceDDD::open(): could not read "
                    << filename << " header.";

    if (header.magic == MAGIC)
    {
      m_is_other_endian = false;
    }
    else if (header.magic == swap_long(MAGIC))
    {
      swap_longs(&header, 4);
      m_is_other_endian = true;
    }
    else
    {
      throw IOErr()
        << "DiskImageResourceDDD::open(): " << filename
        << " has bad magic number (" << header.magic << " != "
        << MAGIC << ").";
    }

    m_bytes_per_pixel = header.bitsPerElement / BITSPERBYTE;
    m_format.planes = 1;
    m_format.pixel_format = VW_PIXEL_GRAY;
    m_format.cols = header.bytesPerScanline / m_bytes_per_pixel;
    m_format.rows = header.numScanlines;

    switch (header.bitsPerElement)
    {
    case BITSPERBYTE:
      m_format.channel_type = VW_CHANNEL_UINT8;
      break;
    case sizeof(short) * BITSPERBYTE:
      // If you think you want/need unsigned ints here talk to LJE before changing!
      //       m_format.channel_type = VW_CHANNEL_UINT16;
       m_format.channel_type = VW_CHANNEL_INT16;
      break;
    default:
      throw IOErr() << "DiskImageResourceDDD::open(): unsupported pixel size ("
                    << header.bitsPerElement << " bits) in " << filename
                    << ".";
    }

    // Put the data into an associative contain (std::map).  Key/value
    // pairs are located by searching for strings seperated by the
    // equals sign "=".
    parse_ddd_header(header.label);

    // Close the file
    image_file.close();
  }

  /// Read the disk image into the given buffer.
  void
  DiskImageResourceDDD::read(ImageBuffer const& dest, BBox2i const& bbox) const
  {
    ifstream image_file(m_filename.c_str(), ios::in | ios::binary);

    if (image_file.bad())
      throw IOErr() << "  DiskImageResourceDDD::read(): \"" << m_filename
                    << "\" is not yet open.";

    // Read the pixel data from the file.
    unsigned int total_pixels = (bbox.width() * bbox.height() *
                                 m_format.planes);
    uint8* image_data = new uint8[total_pixels * m_bytes_per_pixel];

    for (int line = 0; line < bbox.height(); ++line)
    {
      int file_offset =
        IMAGE_HEADER_LENGTH + (((bbox.min().y() + line) * m_format.cols +
                                bbox.min().x()) * m_bytes_per_pixel);
      int image_data_offset =
        (line * bbox.width() + bbox.min().x()) * m_bytes_per_pixel;

      // Set the file offset to the position of the first image
      // byte... the header length is always the same for DDD images.
      image_file.seekg(file_offset, ios::beg);

      // Read one scanline of the bounding box into memory.
      image_file.read((char *) (image_data + image_data_offset),
                      m_bytes_per_pixel * bbox.width());

      if (image_file.bad())
        throw IOErr() << "DiskImageResourceDDD::read():"
          " An unrecoverable error occurred while reading the image data.";
    }


    // DDD images are always big-endian, swap bytes if this is a
    // little-endian system
    if (m_is_other_endian && m_bytes_per_pixel == sizeof(uint16))
      swap_shorts(image_data, total_pixels);

    // Create generic image buffer from the DDD data.
    ImageBuffer src;
    src.data = image_data;
    src.format = m_format;
    src.format.cols = bbox.width();
    src.format.rows = bbox.height();
    src.cstride = m_bytes_per_pixel;
    src.rstride = m_bytes_per_pixel * bbox.width();
    src.pstride = m_bytes_per_pixel * bbox.width() * bbox.height();

    convert(dest, src);

    delete[] image_data;
    image_file.close();
  }

  // Write the given buffer into the disk image.
  void
  DiskImageResourceDDD::write(ImageBuffer const& src, BBox2i const& bbox)
  {
    throw NoImplErr() <<
      "The DDD driver does not yet support creation of DDD files";
  }

  // A FileIO hook to open a file for reading
  DiskImageResource*
  DiskImageResourceDDD::construct_open(std::string const& filename)
  {
    return new DiskImageResourceDDD(filename);
  }

  // A FileIO hook to open a file for writing
  DiskImageResource*
  DiskImageResourceDDD::construct_create(std::string const& filename,
                                         ImageFormat const& format)
  {
    return new DiskImageResourceDDD(filename, format);
  }
}
