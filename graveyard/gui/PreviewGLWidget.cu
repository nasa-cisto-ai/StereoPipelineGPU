#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <QtGui>

// VW
#include <vw/Image.h>
#include <vw/FileIO.h>
using namespace vw;

#include "gui/PreviewGLWidget.h"


// --------------------------------------------------------------
//                       GLSL DEBUGGING
// --------------------------------------------------------------

void printShaderInfoLog(GLuint obj)
{
  int infologLength = 0;
  int charsWritten  = 0;
  char *infoLog;

  glGetShaderiv(obj, GL_INFO_LOG_LENGTH,&infologLength);

  if (infologLength > 0)
    {
      infoLog = (char *)malloc(infologLength);
      glGetShaderInfoLog(obj, infologLength, &charsWritten, infoLog);
      std::ostringstream err;
      err << "<h4>An error occurred while compiling the GLSL shader:</h4><p><h5><tt>" << infoLog << "</tt></h5>";
      QMessageBox::critical(0, "GLSL Shader Error", 
                            err.str().c_str());
      free(infoLog);
    }
}

void printProgramInfoLog(GLuint obj)
{
  int infologLength = 0;
  int charsWritten  = 0;
  char *infoLog;

  glGetProgramiv(obj, GL_INFO_LOG_LENGTH,&infologLength);

  if (infologLength > 0)
    {
      infoLog = (char *)malloc(infologLength);
      glGetProgramInfoLog(obj, infologLength, &charsWritten, infoLog);
      std::ostringstream err;
      err << "<h4>An error occurred while linking the GLSL program:</h4><p><h5><tt>" << infoLog << "</tt></h5>";
      QMessageBox::critical(0, "GLSL Program Error", 
                            err.str().c_str());
      printf("%s\n",infoLog);
      free(infoLog);
    }
}

// --------------------------------------------------------------
//               PreviewGLWidget Public Methods
// --------------------------------------------------------------
PreviewGLWidget::~PreviewGLWidget() {
  makeCurrent();
  glDeleteTextures(1,&m_texture);
}

void PreviewGLWidget::sizeToFit() {
  float aspect = float(m_viewport_width) / m_viewport_height;
  int maxdim = std::max(m_image.cols(),m_image.rows());
  if (m_image.cols() > m_image.rows()) {
    float width = maxdim;
    float height = maxdim/aspect;
    float extra = height - m_image.rows();
    m_current_viewport = BBox2(Vector2(0.0, -extra/2), 
                                Vector2(width, height-extra/2));
  } else {
    float width = maxdim*aspect;
    float height = maxdim;
    float extra = width - m_image.cols();
    m_current_viewport = BBox2(Vector2(-extra/2, 0.0), 
                                Vector2(width-extra/2, height));
  }
  update();
}

void PreviewGLWidget::zoom(float scale) {
  float mid_x = m_current_viewport.min().x() + m_current_viewport.width()/2;
  float mid_y = m_current_viewport.min().y() + m_current_viewport.height()/2;
  
  // Check to make sure we haven't hit our zoom limits...
  if (m_current_viewport.width()/scale > 1.0 && 
      m_current_viewport.height()/scale > 1.0 &&
      m_current_viewport.width()/scale < 4*m_image.cols() && 
      m_current_viewport.height()/scale < 4*m_image.rows()) {
    m_current_viewport.min().x() = (m_current_viewport.min().x() - mid_x) / scale + mid_x;
    m_current_viewport.max().x() = (m_current_viewport.max().x() - mid_x) / scale + mid_x;
    m_current_viewport.min().y() = (m_current_viewport.min().y() - mid_y) / scale + mid_y;
    m_current_viewport.max().y() = (m_current_viewport.max().y() - mid_y) / scale + mid_y;
    update();
  }
  m_show_legend = false;
}

void PreviewGLWidget::normalizeImage() {
  m_offset = -m_image_min;
  m_gain = 1/(m_image_max-m_image_min);
  update();
}

// --------------------------------------------------------------
//             PreviewGLWidget Private Methods
// --------------------------------------------------------------

void PreviewGLWidget::drawImage() {

  // Make this context current, and store the current OpenGL state
  // before we start to modify it.
  makeCurrent();
  glPushAttrib(GL_ALL_ATTRIB_BITS);
  
  // Activate our GLSL fragment program and set up the uniform
  // variables in the shader
  glUseProgram(m_glsl_program);
  GLint gain_loc = glGetUniformLocation(m_glsl_program,"gain");
  glUniform1f(gain_loc,m_gain);
  GLint offset_loc = glGetUniformLocation(m_glsl_program,"offset");
  glUniform1f(offset_loc,m_offset);
  GLint gamma_loc = glGetUniformLocation(m_glsl_program,"gamma");
  glUniform1f(gamma_loc,m_gamma);
  GLint display_channel_loc = glGetUniformLocation(m_glsl_program,"display_channel");
  glUniform1i(display_channel_loc,m_display_channel);

  // Set the background color and viewport.
  qglClearColor(QColor(0, 25, 50)); // Bluish-green background
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glViewport(0,0,m_viewport_width,m_viewport_height);

  // Set up the orthographic view of the scene.  The exact extent of
  // the view onto the scene depends on the current panning and zoom
  // in the UI.
  glMatrixMode(GL_PROJECTION);
  glPushMatrix();
  glLoadIdentity();
  glOrtho(m_current_viewport.min().x(), m_current_viewport.max().x(), 
          -m_current_viewport.max().y(), -m_current_viewport.min().y(),
          -1.0, 1.0);

  // Set up the modelview matrix, and bind the image as the texture we
  // are about to use.
  glMatrixMode(GL_MODELVIEW);
  glPushMatrix();
  glLoadIdentity();
  if (m_draw_texture) {
    glEnable( GL_TEXTURE_2D );
    glBindTexture( GL_TEXTURE_2D, m_texture );
  }
  if (m_bilinear_filter) {
    // When the texture area is small, bilinear filter the closest mipmap
    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST_MIPMAP_LINEAR );
    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR );
  } else {
    // When the texture area is small, pick the nearest neighbor in the closest mipmap
    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST_MIPMAP_NEAREST );
    glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST );
  }


  // Draw the rectangle onto which we will draw the image as a
  // texture.
  qglColor(Qt::white);
  glBegin(GL_QUADS);
  glTexCoord2d( 0.0 , 0.0); 
  glVertex2d( 0.0 , 0.0);
  glTexCoord2d( 0.0 , float(m_image.rows()) / m_image.rows() ); 
  glVertex2d( 0.0 , -(m_image.rows()) );
  glTexCoord2d( float(m_image.cols()) / m_image.cols() , float(m_image.rows()) / m_image.rows() ); 
  glVertex2d( m_image.cols() , -(m_image.rows()) );
  glTexCoord2d( float(m_image.cols()) / m_image.cols() , 0.0 ); 
  glVertex2d( m_image.cols() , 0.0 );
  glEnd();

  // Disable texture mapping and GLSL shaders
  glDisable( GL_TEXTURE_2D );
  glUseProgram(0);

  // Draw crosshairs
  glLineWidth(1.0);
  for (unsigned i = 0; i < m_crosshairs.size(); ++i) {
    Vector3 color = m_crosshairs[i].color();
    glColor3f(color[0], color[1], color[2]);
    glBegin(GL_LINES);
    std::list<Vector2>::const_iterator iter = m_crosshairs[i].points().begin();
    while (iter != m_crosshairs[i].points().end() ) {
      Vector2 point = *iter;
      glVertex2d( point[0]-3 , -point[1]);
      glVertex2d( point[0]+3 , -point[1]);
      glVertex2d( point[0], -point[1]-3);
      glVertex2d( point[0], -point[1]+3);
      ++iter;
    }
    glEnd();
  }    
  
  // Restore the previous OpenGL state so that we don't trample on the
  // QPainter elements of the window.
  glMatrixMode(GL_MODELVIEW);
  glPopMatrix();
  glMatrixMode(GL_PROJECTION);
  glPopMatrix();
  glPopAttrib();
}

void PreviewGLWidget::drawLegend(QPainter* painter) {

  // Extract the value for the pixel currently under the mouse
  PixelRGB<float32> pix_value;
  if (currentImagePos.x() >= 0 && currentImagePos.x() < m_image.cols() &&
      currentImagePos.y() >= 0 && currentImagePos.y() < m_image.rows()) {
    pix_value = m_image(currentImagePos.x(), currentImagePos.y());
  }

  const int Margin = 11;
  const int Padding = 6;

  QTextDocument textDocument;
  textDocument.setDefaultStyleSheet("* { color: #00FF00; font-family: courier, serif }");
  std::ostringstream legend_text;
  legend_text << "<h5 align=\"right\">" << m_legend_status << "<br>"
              << pix_value << "<br>" << " @ " << currentImagePos.x() << " " << currentImagePos.y() << "<br>"
              << "Range: " << m_image_min << " | " << m_offset << " " 
                           << (m_offset + 1/m_gain) << " | " << m_image_max << "</h5>";
  textDocument.setHtml(legend_text.str().c_str());
  textDocument.setTextWidth(textDocument.size().width());
  
  QRect rect(QPoint(0,0), textDocument.size().toSize()
             + QSize(2 * Padding, 2 * Padding));
  painter->translate(width() - rect.width() - Margin,
                    height() - rect.height() - Margin);
  //   painter->setPen(QColor(255, 239, 239));
  //   painter->drawRect(rect);
  painter->translate(Padding, Padding);
  textDocument.drawContents(painter);
}

void PreviewGLWidget::updateCurrentMousePosition() {
  float x_loc = m_current_viewport.min().x() + m_current_viewport.width() * float(lastPos.x()) / m_viewport_width;
  float y_loc = m_current_viewport.min().y() + m_current_viewport.height() * float(lastPos.y()) / m_viewport_height;
  currentImagePos = QPoint(x_loc,y_loc);
}


// --------------------------------------------------------------
//             PreviewGLWidget Setup Methods
// --------------------------------------------------------------
void PreviewGLWidget::setupPreviewGLWidget() {
  setFormat(QGLFormat(QGL::DoubleBuffer | QGL::DepthBuffer));

  // Set some reasonable defaults
  m_draw_texture = true;
  m_show_legend = false;
  m_bilinear_filter = true;
  m_use_colormap = false;
  m_adjust_mode = TransformAdjustment;
  m_display_channel = DisplayRGBA;
  
  // Set up shader parameters
  m_gain = 1.0;
  m_offset = 0.0;
  m_gamma = 1.0;
  
  // Allocate the opengl texture
  glGenTextures(1,&m_texture);

  // Set mouse tracking
  this->setMouseTracking(true);

  // Set the size policy that the widget can grow or shrink and still
  // be useful.
  this->setSizePolicy(QSizePolicy::Expanding, QSizePolicy::Expanding);
}

void PreviewGLWidget::initializeGL() {  
  makeCurrent();

  glShadeModel(GL_FLAT);

  // Set up the texture mode to replace (rather than blend...)
  glBindTexture(GL_TEXTURE_2D, m_texture);
  glTexEnvf( GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE );

  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_GENERATE_MIPMAP, GL_TRUE); 

  // Copy the texture data over into texture memory.
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, m_image.cols(), m_image.rows(), 0, 
               GL_RGB, GL_FLOAT, &(m_image(0,0)) );
  
  std::ifstream input_file("/Users/mbroxton/projects/StereoPipeline/trunk/src/gui/PreviewGLWidget.frag");
  if (!input_file.is_open())
    vw_throw(IOErr() << "Could not open GLSL shader file.");
  std::string fragment_prog, line;
  while (!input_file.eof()) {
    getline(input_file,line);
    fragment_prog += (line + "\n");
  }
  input_file.close();
  const char* fragment_prog_ptr = fragment_prog.c_str();

  // For debugging:
  //  std::cout << "***\n" << fragment_prog << "***\n";
  
  GLuint m_fragment_shader = glCreateShader(GL_FRAGMENT_SHADER);
  glShaderSource(m_fragment_shader, 1, &fragment_prog_ptr, NULL);
  glCompileShader(m_fragment_shader);
  printShaderInfoLog(m_fragment_shader);
  
  m_glsl_program = glCreateProgram();
  glAttachShader(m_glsl_program, m_fragment_shader);
  glLinkProgram(m_glsl_program);
  printProgramInfoLog(m_glsl_program);
}

void PreviewGLWidget::resizeGL(int width, int height) {
  m_viewport_width = width;
  m_viewport_height = height;
  sizeToFit();
}


// --------------------------------------------------------------
//             PreviewGLWidget Event Handlers
// --------------------------------------------------------------

void PreviewGLWidget::paintEvent(QPaintEvent * /* event */) { 
  QPainter painter(this);
  drawImage();
  if (m_show_legend)
    drawLegend(&painter);
}

void PreviewGLWidget::mousePressEvent(QMouseEvent *event) { 
  m_show_legend = true;
  grabKeyboard();
  lastPos = event->pos();
  updateCurrentMousePosition();
}

void PreviewGLWidget::mouseMoveEvent(QMouseEvent *event) {
  float ticks;

  // Left mouse button moves the image around
  if (event->buttons() & Qt::LeftButton) {
    float x_diff = float(event->x() - lastPos.x()) / m_viewport_width;
    float y_diff = float(event->y() - lastPos.y()) / m_viewport_height;

    std::ostringstream s; 
    switch (m_adjust_mode) {

    case TransformAdjustment:
      m_current_viewport.min().x() -= x_diff * m_current_viewport.width();
      m_current_viewport.min().y() -= y_diff * m_current_viewport.height();
      m_current_viewport.max().x() -= x_diff * m_current_viewport.width();
      m_current_viewport.max().y() -= y_diff * m_current_viewport.height();
      break;

    case GainAdjustment:
      // The number '5' below adjust the sensitivity.
      ticks = pow(2, 5 * x_diff * (m_image_max-m_image_min));
      if (m_gain * ticks > 1e-8 && m_gain * ticks < 1e8)
        m_gain *= ticks;
      s << "Gain: " << m_gain << "   " << x_diff << "   " << ticks << "\n";
      m_legend_status = s.str();
      break;

    case OffsetAdjustment:
      m_offset += x_diff * (m_image_max - m_image_min);
      s << "Offset: " << m_offset << "\n";
      m_legend_status = s.str();
      break;

    case GammaAdjustment:
      // The number '5.0' below adjust the sensitivity.
      ticks = pow(2, x_diff * 5.0);
      if (m_gamma * ticks > 0.01 && m_gamma * ticks < 10.0)
        m_gamma *= ticks;
      s << "Gamma: " << m_gamma << "\n";
      m_legend_status = s.str();
      break;
    }

  } else if (event->buttons() & Qt::RightButton) {
    m_gain += GLfloat(event->x() - lastPos.x()) / m_viewport_width *10;
  } 

  // Regardless, we store the current position for the text legend.
  lastPos = event->pos();
  updateCurrentMousePosition();
  update();
}

void PreviewGLWidget::mouseDoubleClickEvent(QMouseEvent * /*event*/) {
  m_draw_texture = !m_draw_texture;
  update();
}

void PreviewGLWidget::wheelEvent(QWheelEvent *event) {
  int num_degrees = event->delta() / 8;
  float num_ticks = num_degrees / 15;

  float scale = pow(2,num_ticks/5);
  zoom(scale);

  m_show_legend = true;
  grabKeyboard();
  lastPos = event->pos();
  updateCurrentMousePosition();
}


void PreviewGLWidget::enterEvent(QEvent */*event*/) {
  m_show_legend = true;
  grabKeyboard();
  update();
}

void PreviewGLWidget::leaveEvent(QEvent */*event*/) {
  m_show_legend = false;
  releaseKeyboard();
  update();
}

void PreviewGLWidget::keyPressEvent(QKeyEvent *event) {

  std::ostringstream s; 
  
  switch (event->key()) {
  case Qt::Key_Plus:   // Zoom inxo
    zoom(2.0);
    break;
  case Qt::Key_Minus:  // Zoom out
    zoom(0.5);
    break;
  case Qt::Key_F:  // Size to fit
    sizeToFit();
    break;
  case Qt::Key_N:  // Toggle bilinear/nearest neighbor interp
    m_bilinear_filter = !m_bilinear_filter;
    update();
    break;
  case Qt::Key_C:  // Activate colormap
    m_use_colormap = !m_use_colormap;
    update();
    break;
  case Qt::Key_R:  // Normalize the image
    normalizeImage();
    update();
    break;
  case Qt::Key_G:  // Gain adjustment mode
    if (m_adjust_mode == GainAdjustment) {
      m_adjust_mode = TransformAdjustment;
      m_legend_status = "";
    } else {
      m_adjust_mode = GainAdjustment;
      s << "Gain: " << m_gain;
      m_legend_status = s.str();
    }
    update();
    break;
  case Qt::Key_O:  // Offset adjustment mode
    if (m_adjust_mode == OffsetAdjustment) {
      m_adjust_mode = TransformAdjustment;
      m_legend_status = "";
    } else {
      m_adjust_mode = OffsetAdjustment;
      s << "Offset: " << m_offset;
      m_legend_status = s.str();
    }
    update();
    break;
  case Qt::Key_V:  // Gamma adjustment mode
    if (m_adjust_mode == GammaAdjustment) {
      m_adjust_mode = TransformAdjustment;
      m_legend_status = "";
    } else {
      m_adjust_mode = GammaAdjustment;
      s << "Gamma: " << m_gamma;
      m_legend_status = s.str();
    }
    update();
    break;
  case Qt::Key_1:  // Normalize the image
    m_display_channel = DisplayR;
    update();
    break;
  case Qt::Key_2:  // Normalize the image
    m_display_channel = DisplayG;
    update();
    break;
  case Qt::Key_3:  // Normalize the image
    m_display_channel = DisplayB;
    update();
    break;
  case Qt::Key_4:  // Normalize the image
    m_display_channel = DisplayA;
    update();
    break;
  case Qt::Key_0:  // Normalize the image
    m_display_channel = DisplayRGBA;
    update();
    break;
  default: 
    QWidget::keyPressEvent(event);
  }
}

void PreviewGLWidget::add_crosshairs(std::list<Vector2> const& points, Vector3 const& color) {
  m_crosshairs.push_back(PointList(points, color));
  update();
}

void PreviewGLWidget::clear_crosshairs() {
  m_crosshairs.clear(); 
  update();
}



