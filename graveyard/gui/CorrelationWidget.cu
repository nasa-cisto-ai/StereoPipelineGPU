#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <QtGui>

#include "CorrelationWidget.h"

CorrelationWidget::CorrelationWidget(StereoGuiSession *cs, QWidget *parent) : QWidget(parent) {
  this->blockSignals(true);

  m_cs = cs;

  m_xImagePreview = new PreviewGLWidget(this);
  m_yImagePreview = new PreviewGLWidget(this);
  m_leftImagePreview = new PreviewGLWidget(this);

  QTabWidget *previewTab = new QTabWidget;
  previewTab->addTab(m_xImagePreview, "X Disparity");
  previewTab->addTab(m_yImagePreview, "Y Disparity");
  previewTab->addTab(m_leftImagePreview, "Left Image");

  QVBoxLayout *mainLayout = new QVBoxLayout;
  mainLayout->addWidget(previewTab);
  mainLayout->addWidget(genSettingsBox("Search Window Preview Settings"));

  this->setLayout(mainLayout);

  //  m_correlateThread = new QCorrelateThread(this);

//   connect(m_correlateThread, SIGNAL(correlationFinished()), this, SLOT(correlationFinished()));
//   connect(m_correlateThread, SIGNAL(progressUpdate(int)), m_progressBar, SLOT(setValue(int)));
  
//   connect(m_cancelButton, SIGNAL(clicked()), m_correlateThread, SLOT(abortCorrelation()));
  connect(m_cancelButton, SIGNAL(clicked()), m_progressBar, SLOT(reset()));

  connect(m_doCorrelateButton, SIGNAL(clicked()), this, SLOT(doCorrelate()));

  connect(m_xImagePreview, SIGNAL(imageClicked(int, int)), this, SLOT(imageClicked(int, int)));
  connect(m_yImagePreview, SIGNAL(imageClicked(int, int)), this, SLOT(imageClicked(int, int)));
  connect(m_leftImagePreview, SIGNAL(imageClicked(int, int)), this, SLOT(imageClicked(int, int)));

  connect(previewTab, SIGNAL(currentChanged(int)), m_xImagePreview, SLOT(fitToWindow()));
  connect(previewTab, SIGNAL(currentChanged(int)), m_yImagePreview, SLOT(fitToWindow()));
  connect(previewTab, SIGNAL(currentChanged(int)), m_leftImagePreview, SLOT(fitToWindow()));
}

QGroupBox *CorrelationWidget::genSettingsBox(QString const& name) {
  m_doCorrelateButton = new QPushButton("Correlate!!");

  QVBoxLayout *layout = new QVBoxLayout;
  layout->addWidget(m_doCorrelateButton);
  layout->addLayout(genProgressLayout());

  QGroupBox *box = new QGroupBox(name);
  box->setSizePolicy(QSizePolicy::Preferred, QSizePolicy::Maximum);

  box->setLayout(layout);

  return box;
}

QHBoxLayout *CorrelationWidget::genProgressLayout() {
  m_progressBar = new QProgressBar;
  m_progressBar->setRange(0, 100);
  m_progressBar->setValue(0);

  m_cancelButton = new QPushButton("Cancel");

  QHBoxLayout *layout = new QHBoxLayout;
  layout->addWidget(m_progressBar);
  layout->addWidget(m_cancelButton);

  return layout;
}

void CorrelationWidget::doCorrelate() {
  if (m_cs->hasBothImagesLoaded()) {
    //    m_correlateThread->doCorrelate(m_cs->getCostFunctionObject(), m_cs->searchWindow());
  }
}

void CorrelationWidget::updatePreview() {
//   m_xImagePreview->setCrosshairPosition(m_cs->searchWindowPreviewPoint());
//   m_xImagePreview->setCrosshairEnabled(m_cs->crosshairEnabled());
//   m_xImagePreview->updatePreview();
  
//   m_yImagePreview->setCrosshairPosition(m_cs->searchWindowPreviewPoint());
//   m_yImagePreview->setCrosshairEnabled(m_cs->crosshairEnabled());
//   m_yImagePreview->updatePreview();

//   m_leftImagePreview->setCrosshairPosition(m_cs->searchWindowPreviewPoint());
//   m_leftImagePreview->setCrosshairEnabled(m_cs->crosshairEnabled());
//   m_leftImagePreview->updatePreview();
}

void CorrelationWidget::imageClicked(int x, int y) {
  vw::Vector2i loc(x, y);
  
  if (loc == m_cs->searchWindowPreviewPoint() && m_cs->crosshairEnabled()) {
    m_cs->setCrosshairEnabled(false);
  }
  else {
    m_cs->setCrosshairEnabled(true);
  }

  m_cs->setSearchWindowPreviewPoint(vw::Vector2i(x, y));

  updatePreview();
}

void CorrelationWidget::updateWidgets() {
  //  m_leftImagePreview->setImage(m_cs->leftImage(), true);
  updatePreview();
}

void CorrelationWidget::correlationFinished() {
  //  vw::ImageView<vw::PixelDisparity<vw::float32> > disparity_map = m_correlateThread->result();

//   vw::ImageView<vw::float32> xResult = clamp(select_channel(disparity_map, 0), m_cs->searchWindow().min().x(), m_cs->searchWindow().max().x());
//   vw::ImageView<vw::float32> yResult = clamp(select_channel(disparity_map, 1), m_cs->searchWindow().min().y(), m_cs->searchWindow().max().y());

//   m_xImagePreview->setImage(xResult, true);
//   m_yImagePreview->setImage(yResult, true);

//   m_xImagePreview->fitToWindow();
//   m_yImagePreview->fitToWindow();
//   m_leftImagePreview->fitToWindow();

//   m_cs->setDisparityMap(disparity_map);
 
  updatePreview();
}
