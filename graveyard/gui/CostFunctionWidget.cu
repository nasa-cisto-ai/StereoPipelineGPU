#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <QtGui>
#include <cmath>

#include "gui/CostFunctionWidget.h"
#include "gui/QCompatFormLayout.h"
#include "gui/PreviewGLWidget.h"

CostFunctionWidget::CostFunctionWidget(QWidget *parent) : QWidget(parent) {
  this->blockSignals(true);

  m_imagePreview = new PreviewGLWidget(this);
  m_leftImagePreview = new PreviewGLWidget(this);

  QTabWidget *previewTab = new QTabWidget;
  previewTab->addTab(m_imagePreview, "Cost Landscape");
  previewTab->addTab(m_leftImagePreview, "Left Image");

  QVBoxLayout *mainLayout = new QVBoxLayout;
  mainLayout->addWidget(previewTab);
  mainLayout->addWidget(genSettingsBox("Cost Function Preview Settings"));

  this->setLayout(mainLayout);

  connect(m_xDisparitySlider, SIGNAL(valueChanged(int)), m_xDisparitySpin, SLOT(setValue(int)));
  connect(m_yDisparitySlider, SIGNAL(valueChanged(int)), m_yDisparitySpin, SLOT(setValue(int)));
  connect(m_xDisparitySpin, SIGNAL(valueChanged(int)), m_xDisparitySlider, SLOT(setValue(int)));
  connect(m_yDisparitySpin, SIGNAL(valueChanged(int)), m_yDisparitySlider, SLOT(setValue(int)));
  
//   connect(m_xDisparitySpin, SIGNAL(valueChanged(int)), m_cs, SLOT(setXDisparityPreview(int)));
//   connect(m_yDisparitySpin, SIGNAL(valueChanged(int)), m_cs, SLOT(setYDisparityPreview(int)));

//   connect(m_cs, SIGNAL(xDisparityPreviewChanged(int)), m_xDisparitySpin, SLOT(setValue(int)));
//   connect(m_cs, SIGNAL(yDisparityPreviewChanged(int)), m_yDisparitySpin, SLOT(setValue(int)));

//   connect(m_cs, SIGNAL(disparityPreviewChanged(int, int)), this, SLOT(recalculateCost()));

//   connect(m_cs, SIGNAL(costFunctionObjectChanged(boost::shared_ptr<vw::stereo::CostFunction<vw::float32> >)), this, SLOT(recalculateCost()));
  
  connect(m_imagePreview, SIGNAL(imageClicked(int, int)), this, SLOT(imageClicked(int, int)));
  connect(m_leftImagePreview, SIGNAL(imageClicked(int, int)), this, SLOT(imageClicked(int, int)));

  connect(previewTab, SIGNAL(currentChanged(int)), m_imagePreview, SLOT(fitToWindow()));
  connect(previewTab, SIGNAL(currentChanged(int)), m_leftImagePreview, SLOT(fitToWindow()));
}

QGroupBox *CostFunctionWidget::genSettingsBox(QString const& name) {
  QCompatFormLayout *layout = new QCompatFormLayout;
  
  QLabel *xDisparityLabel = new QLabel("X Disparity:");
  layout->addRow(xDisparityLabel, genXDisparityKnobs());
  
  QLabel *yDisparityLabel = new QLabel("Y Disparity:");
  layout->addRow(yDisparityLabel, genYDisparityKnobs());
  
  QGroupBox *box =  new QGroupBox(name);
  box->setSizePolicy(QSizePolicy::Preferred, QSizePolicy::Maximum);
  box->setLayout(layout);
  return box;
}

QHBoxLayout *CostFunctionWidget::genXDisparityKnobs() {
  m_xDisparitySpin = new QSpinBox;
  m_xDisparitySpin->setValue(0);
  m_xDisparitySpin->setSizePolicy(QSizePolicy::Maximum, QSizePolicy::Fixed);
  
  m_xDisparitySlider = new QSlider(Qt::Horizontal);
  m_xDisparitySlider->setValue(0);
  m_xDisparitySlider->setPageStep(1);

  QHBoxLayout *layout = new QHBoxLayout;
  layout->addWidget(m_xDisparitySlider);
  layout->addWidget(m_xDisparitySpin);
  return layout;
}

QHBoxLayout *CostFunctionWidget::genYDisparityKnobs() {
  m_yDisparitySpin = new QSpinBox;
  m_yDisparitySpin->setValue(0);
  m_yDisparitySpin->setSizePolicy(QSizePolicy::Maximum, QSizePolicy::Fixed);
  
  m_yDisparitySlider = new QSlider(Qt::Horizontal);
  m_yDisparitySlider->setValue(0);
  m_yDisparitySlider->setPageStep(1);

  QHBoxLayout *layout = new QHBoxLayout;
  layout->addWidget(m_yDisparitySlider);
  layout->addWidget(m_yDisparitySpin);
  return layout;
}

void CostFunctionWidget::recalculateCost() {
//   if (m_cs->hasBothImagesLoaded()) {
//     int width = m_cs->leftImage().cols();
//     int height = m_cs->leftImage().rows();
//     int dx = m_xDisparitySpin->value();
//     int dy = m_yDisparitySpin->value();

//     vw::BBox2i corr_window((dx < 0) ? (-dx) : 0,
//                            (dy < 0) ? (-dy) : 0,
//                            width - abs(dx),
//                            height - abs(dy));
    
//     vw::ImageView<vw::float32> result(width, height);

//     vw::crop(result, corr_window) = m_cs->getCostFunctionObject()->calculate(corr_window, vw::Vector2i(dx, dy));

//     m_imagePreview->setImage(result, true);

//     updatePreview();
//   }
}

void CostFunctionWidget::updatePreview() {
//   m_imagePreview->setCrosshairPosition(m_cs->searchWindowPreviewPoint());
//   m_imagePreview->setCrosshairEnabled(m_cs->crosshairEnabled());
    
//   m_imagePreview->updatePreview(); 

//   m_leftImagePreview->setCrosshairPosition(m_cs->searchWindowPreviewPoint());
//   m_leftImagePreview->setCrosshairEnabled(m_cs->crosshairEnabled());

//   m_leftImagePreview->updatePreview();
}

void CostFunctionWidget::imageClicked(int x, int y) {
//   vw::Vector2i loc(x, y);
  
//   if (loc == m_cs->searchWindowPreviewPoint() && m_cs->crosshairEnabled()) {
//     m_cs->setCrosshairEnabled(false);
//   }
//   else {
//     m_cs->setCrosshairEnabled(true);
//   }

//   m_cs->setSearchWindowPreviewPoint(vw::Vector2i(x, y));

//   updatePreview();
}

void CostFunctionWidget::updateWidgets() {
//   int width = m_cs->leftImage().cols();
//   int height = m_cs->leftImage().rows();

//   m_xDisparitySpin->setRange(-width, width);
//   m_yDisparitySpin->setRange(-height, height);
//   m_xDisparitySlider->setRange(-width, width);
//   m_yDisparitySlider->setRange(-height, height);

//   m_leftImagePreview->setImage(m_cs->leftImage());
  
//   recalculateCost();

//   m_imagePreview->fitToWindow();
}
