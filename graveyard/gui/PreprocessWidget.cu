#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <QtGui>
#include <QVBoxLayout>
#include <QHBoxLayout>

#include <vw/Image.h>
#include <vw/FileIO.h>
#include <vw/Stereo.h>

using namespace vw;

#include "gui/QCompatFormLayout.h"
#include "gui/PreprocessWidget.h"
#include "gui/PreviewGLWidget.h"
#include "gui/Noise.h"



template<class ImageT>
ImageView<typename ImageT::pixel_type> box_filter(ImageViewBase<ImageT> const& img,  Vector2i const& kernSize) {
  typedef typename ImageT::pixel_type PixelT;

  ImageView<PixelT> src = img.impl();
    
  ImageView<PixelT> result(src.cols(), src.rows());
    
  Vector<PixelT> cSum(src.cols());

  // Seed the column sum buffer
  for (int x = 0; x < src.cols(); x++) {
    cSum(x) = 0;
    for (int ky = 0; ky < kernSize.y(); ky++) {
      cSum(x) += src(x, ky);
    }
  }
    
  for (int y = 0; y < src.rows() - kernSize.y(); y++) {
    // Seed the row sum
    PixelT rsum = 0;
    for (int i = 0; i < kernSize.x(); i++) {
      rsum += cSum(i);
    }

    for (int x = 0; x < src.cols() - kernSize.x(); x++) {
      result(x + kernSize.x() / 2, y + kernSize.y() / 2) = rsum;
      // Update the row sum
      rsum += cSum(x + kernSize.x()) - cSum(x);
    }

    // Update the column sum
    for (int i = 0; i < src.cols(); i++) {
      cSum(i) += src(i, y + kernSize.y()) - src(i, y);
    }
  }
   
  return result / (kernSize.x() * kernSize.y()); 
}

template <class ImageT>
ImageViewRef<typename ImageT::pixel_type> calc_preproc_filter(ImageViewBase<ImageT> const& image, stereopipeline::PreprocFilterType preproc_filter, double size = 0) {
  switch (preproc_filter) {
  case stereopipeline::NO_PREPROC_FILTER:
    return image.impl();
    break;
  case stereopipeline::LOG_PREPROC_FILTER:
    return laplacian_filter(gaussian_filter(image.impl(), size));
    break;
  case stereopipeline::SLOG_PREPROC_FILTER:
    return threshold(laplacian_filter(gaussian_filter(image.impl(), size)), 0.0);
    break;
  case stereopipeline::GAUSSIAN_PREPROC_FILTER:
    return gaussian_filter(image.impl(), size);
    break;
  case stereopipeline::BOX_PREPROC_FILTER:
    return box_filter(image.impl(), Vector2i(int(size), int(size)));
    break;
  default:
    VW_ASSERT(0, ArgumentErr() << "Unrecognized Preprocessing Filter");
    return ImageViewRef<typename ImageT::pixel_type>();
  }
}


PreprocessWidget::PreprocessWidget(QString const& name, QWidget *parent) : QWidget(parent) {
  m_glPreview = new PreviewGLWidget(this);

  QVBoxLayout *mainLayout = new QVBoxLayout;
  mainLayout->addWidget(m_glPreview);
  mainLayout->addWidget(genSettingsBox(name + " Settings"));
  this->setLayout(mainLayout);

  connect(m_fileBrowseButton, SIGNAL(clicked()), this, SLOT(fileBrowseButtonClicked()));
  connect(m_fileNameEdit, SIGNAL(returnPressed()), this, SLOT(loadImage()));
  
  connect(m_filterTypeBox, SIGNAL(currentIndexChanged(QString)), this, SLOT(updateImage())); 
  connect(m_gaussNoiseSpin, SIGNAL(valueChanged(double)), this, SLOT(updateImage()));
  connect(m_spNoiseSpin, SIGNAL(valueChanged(double)), this, SLOT(updateImage()));
  connect(m_gammaSpin, SIGNAL(valueChanged(double)), this, SLOT(updateImage()));
  connect(m_filterSizeSpin, SIGNAL(valueChanged(double)), this, SLOT(updateImage()));
}

QGroupBox *PreprocessWidget::genSettingsBox(QString const& name) {
  QLabel *gaussLabel = new QLabel("Gaussian Noise:");
  m_gaussNoiseSpin = new QDoubleSpinBox;
  m_gaussNoiseSpin->setValue(0.0);
  m_gaussNoiseSpin->setRange(0.0, 3.0);
  m_gaussNoiseSpin->setSingleStep(0.1);

  QLabel *spLabel = new QLabel("Salt and Pepper Noise:");
  m_spNoiseSpin = new QDoubleSpinBox;
  m_spNoiseSpin->setValue(0.0);
  m_spNoiseSpin->setRange(0.0, 1.0);
  m_spNoiseSpin->setSingleStep(0.1);

  QLabel *gammaLabel = new QLabel("Gamma:");
  m_gammaSpin = new QDoubleSpinBox;
  m_gammaSpin->setValue(1.0);
  m_gammaSpin->setRange(0.0, 10.0);
  m_gammaSpin->setSingleStep(0.1);

  QLabel *filterTypeLabel = new QLabel("Filter Type:");
  m_filterTypeBox = new PreprocessComboBox(this);

  QLabel *filterSizeLabel = new QLabel("Filter Kernel Size:");
  m_filterSizeSpin = new QDoubleSpinBox;
  m_filterSizeSpin->setValue(1.5);
  m_filterSizeSpin->setRange(0.0, 1000);
  m_filterSizeSpin->setSingleStep(0.1);

  QCompatFormLayout *layout = new QCompatFormLayout;

  layout->addRow(genFileOpenLayout());
  layout->addRow(gaussLabel, m_gaussNoiseSpin);
  layout->addRow(spLabel, m_spNoiseSpin);
  layout->addRow(gammaLabel, m_gammaSpin);
  layout->addRow(filterTypeLabel, m_filterTypeBox);
  layout->addRow(filterSizeLabel, m_filterSizeSpin);

  QGroupBox *box = new QGroupBox(name);
  box->setSizePolicy(QSizePolicy::Preferred, QSizePolicy::Maximum);

  box->setLayout(layout);

  return box;
}

QHBoxLayout *PreprocessWidget::genFileOpenLayout() {
  QLabel *fileLabel = new QLabel("Filename: ");

  m_fileNameEdit = new QLineEdit;
  m_fileBrowseButton = new QPushButton("Browse");
  QHBoxLayout *layout = new QHBoxLayout;
  layout->addWidget(fileLabel);
  layout->addWidget(m_fileNameEdit);
  layout->addWidget(m_fileBrowseButton);
  return layout;
}

void PreprocessWidget::updateImage() {
  if (hasImageLoaded()) {
    vw::ImageViewRef<PixelRGB<vw::float32> > pipeline = vw::channel_cast<vw::float32>(m_inputImage);

  //   if (m_gaussNoiseSpin->value() != 0) {
//       pipeline = gaussian_noise(pipeline, m_gaussNoiseSpin->value());
//     }

    if (m_spNoiseSpin->value() != 0) {
      pipeline = salt_pepper_noise(pipeline, m_spNoiseSpin->value());
    }

    if (m_gammaSpin->value() != 1) {
      pipeline = pow(pipeline, vw::float32(m_gammaSpin->value()));
    }

    // I used to not need raster, and could put pipeline directly into
    // the preproc_filter I wonder why this isn't working anymore
    vw::ImageView<PixelRGB<vw::float32> > raster = pipeline; 
    m_resultImage = calc_preproc_filter(raster, 
                                        m_filterTypeBox->value(), 
                                        m_filterSizeSpin->value());
    m_glPreview->setImage(m_resultImage);
  }  
  
}

void PreprocessWidget::fileBrowseButtonClicked() {
  QString filename = QFileDialog::getOpenFileName(this, "Open...", "", "Images (*.png *.jpg *.tif *.cub *.img)");

  if (filename != "") {
    m_fileNameEdit->setText(filename);
    loadImage();
  }
}

void PreprocessWidget::loadImage() {
  QString filename = m_fileNameEdit->text();

  if (filename != "") {
    try {
      read_image(m_inputImage, filename.toStdString());
      m_fileNameEdit->setText(filename);
      updateImage();
      m_glPreview->sizeToFit();
    }
    catch(vw::Exception& e) {
      QMessageBox::critical(this, "Error opening image", e.what());
    }
  }
}
