#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


#include <QtGui>
#include <QVBoxLayout>
#include <QHBoxLayout>

#include <vw/Image.h>
#include <vw/FileIO.h>
#include <vw/Stereo.h>
using namespace vw;

#include "StereoSettings.h"
#include "gui/StereoGuiSession.h"
#include "gui/AlignmentWidget.h"
#include "gui/PreviewGLWidget.h"
#include "gui/QCompatFormLayout.h"

AlignmentWidget::AlignmentWidget(QWidget *parent) : QWidget(parent) {

  QVBoxLayout *mainLayout = new QVBoxLayout;
  mainLayout->addWidget(generate_previews());
  mainLayout->addWidget(generate_controls());
  this->setLayout(mainLayout);
  
}

QWidget* AlignmentWidget::generate_previews() {

  // Comparison for the two aligned images (in two separate windows)
  m_left_preview = new PreviewGLWidget(this);
  m_right_preview = new PreviewGLWidget(this);
  QHBoxLayout *align_preview_layout = new QHBoxLayout;
  align_preview_layout->addWidget(m_left_preview);
  align_preview_layout->addWidget(m_right_preview);
  QWidget *aligned_preview = new QWidget(this);
  aligned_preview->setLayout(align_preview_layout);

  // Comparison for the two aligned images (in the same window)
  m_compare_preview = new PreviewGLWidget(this);

  // Comparison for the two original images (with interest point overlay)
  m_left_ip_preview = new PreviewGLWidget(this);
  m_right_ip_preview = new PreviewGLWidget(this);
  QHBoxLayout *align_ip_layout = new QHBoxLayout;
  align_ip_layout->addWidget(m_left_ip_preview);
  align_ip_layout->addWidget(m_right_ip_preview);
  QWidget *ip_preview = new QWidget(this);
  ip_preview->setLayout(align_ip_layout);

  // The tab widget allows the user to cycle through the three views
  // above.
  QTabWidget *tab = new QTabWidget(this);
  tab->setTabPosition(QTabWidget::South);
  tab->addTab(aligned_preview, "Aligned");
  tab->addTab(m_compare_preview, "Compare");
  tab->addTab(ip_preview, "Interest Points");

  // Image Previews Group Box
  QHBoxLayout *master_layout = new QHBoxLayout;
  master_layout->addWidget(tab);
  QGroupBox *previews = new QGroupBox("Image Previews");
  previews->setLayout(master_layout);

  // Set up the signal and slot connections
  connect(stereo_gui_session().qobject_ptr(), SIGNAL(left_aligned_image_changed(std::string)), m_left_preview, SLOT( load_image_from_file(std::string) ));
  connect(stereo_gui_session().qobject_ptr(), SIGNAL(right_aligned_image_changed(std::string)), m_right_preview, SLOT( load_image_from_file(std::string) ));

  return previews;
}

QWidget* AlignmentWidget::generate_controls() {

  // Set up feature-based alignment group box
  QVBoxLayout *feature_layout = new QVBoxLayout;
  QPushButton *feature_button = new QPushButton("Align Images");
  feature_button->setSizePolicy(QSizePolicy::Preferred, QSizePolicy::Preferred);
  feature_layout->addWidget(new QLabel("Description of feature based alignment..."));
  feature_layout->addWidget(feature_button);

  QWidget* feature_widget = new QWidget(this);
  feature_widget->setLayout(feature_layout);
  
  // Set up orthoimage alignment group box
  QVBoxLayout *orthoimage_layout = new QVBoxLayout;
  QPushButton *orthoimage_button = new QPushButton("Align Images");
  orthoimage_button->setSizePolicy(QSizePolicy::Preferred, QSizePolicy::Preferred);
  orthoimage_layout->addWidget(new QLabel("Description of orthoimage based alignment..."));
  orthoimage_layout->addWidget(orthoimage_button);

  QWidget *orthoimage_widget = new QWidget(this);
  orthoimage_widget->setLayout(orthoimage_layout);

  // Set up tabe widget for both control boxes
  QTabWidget *controls = new QTabWidget(this);
  controls->setTabPosition(QTabWidget::South); 
  controls->addTab(feature_widget, "Feature-based Alignment");
  controls->addTab(orthoimage_widget, "Orthoimage-based Alignment");
  controls->setSizePolicy(QSizePolicy::Preferred, QSizePolicy::Maximum);

  // Set up the signal and slot connections
  connect(feature_button, SIGNAL(clicked()), this, SLOT( feature_button_clicked() ));
  connect(orthoimage_button, SIGNAL(clicked()), this, SLOT( orthoimage_button_clicked() ));

  return controls;
}


void AlignmentWidget::feature_button_clicked() {
  std::string pre_preprocess_file1, pre_preprocess_file2;
  stereo_gui_session().session()->preprocessing_hook(stereo_gui_session().left_input_image(), 
                                                         stereo_gui_session().right_input_image(),
                                                         pre_preprocess_file1, pre_preprocess_file2);

  std::cout << "\nGenerating image masks..." << std::flush;
  DiskImageView<PixelGray<uint8> > left_rectified_image(pre_preprocess_file1);
  DiskImageView<PixelGray<uint8> > right_rectified_image(pre_preprocess_file2);
  int mask_buffer = std::max(stereo_settings().h_kern, stereo_settings().v_kern);
  ImageViewRef<uint8> Lmask = channel_cast_rescale<uint8>(stereo::disparity::generate_mask(left_rectified_image, mask_buffer));
  ImageViewRef<uint8> Rmask = channel_cast_rescale<uint8>(stereo::disparity::generate_mask(right_rectified_image, mask_buffer));
  std::cout << "Done.\n";
  write_image(stereo_gui_session().output_prefix() + "-lMask.tif", Lmask);
  write_image(stereo_gui_session().output_prefix() + "-rMask.tif", Rmask);
  
  stereo_gui_session().set_left_aligned_image(pre_preprocess_file1);
  stereo_gui_session().set_right_aligned_image(pre_preprocess_file2);
}

void AlignmentWidget::orthoimage_button_clicked() {
  
}
