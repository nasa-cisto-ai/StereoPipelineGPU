#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2009-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NGT platform is licensed under the Apache License, Version 2.0 (the
//  "License"); you may not use this file except in compliance with the
//  License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__


/// \file nff_terrain.cc
///

/************************************************************************/
/*     File: nff_terrain.c                                              */
/*     Date: August 1996                                                */
/*       By: Eric Zbinden                                               */
/* Modified: August 1999, by Larry Edwards                              */
/*      For: NASA Ames Research Center, Intelligent Mechanisms Group    */
/* Function: Main program for the stereo panorama pipeline              */
/*    Links:    stereo.h        stereo.c                                */
/*              filters.h       filters.c                               */
/*              stereo_lib.h    stereolib.c                             */
/*              model_lib.h     modellib.c                              */
/*              stereo.default                                          */
/*                                                                      */
/*    Notes: Stereo.c rebuild a 3D model of the world from a stereo     */
/*           Panorama. St_pan->filtering->disparity_map->range_map:     */
/*              -> dot cloud 3D model                                   */
/*              -> 3D .nff file                                         */
/*                                                                      */
/************************************************************************/

/* The following three includes are for debugging -LJE */
#include <sys/types.h>
#ifdef __APPLE__
#include <malloc/malloc.h>
#include <float.h>                         // for DBL_MAX & FLT_MAX
#else
#include <malloc.h>
#include <values.h>                        // for DBL_MAX & FLT_MAX
#endif
#include <assert.h>
#include <string.h> /* strrchr() */

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// STL include
#include <stdexcept>
#include <vector>

// ASP includes:
#include <asp/asp_config.h>
#include <asp/Tools/nff_terrain.h>

// VW
#include <vw/Core/Exception.h>
#include <vw/Math/Vector.h>

struct PixelCoords
{
  PixelCoords() { row = col = 0; }
  PixelCoords(int x, int y) { row = y; col = x; }
  int PixelIndex(int imageWidth);

  int row,col;
};                      /* xy vector or vertex position */

#if defined(ASP_HAVE_PKG_OPENSCENEGRAPH) && ASP_HAVE_PKG_OPENSCENEGRAPH==1
#include <osg/Geometry>
#include <osg/Material>
#include <osg/Geode>
#include <osg/Texture2D>
#include <osg/Image>
#include <osgDB/WriteFile>
#include <osgDB/ReadFile>
#endif

// STL includes:
#include <iostream>                        // debugging
#include <list>
#include <set>
#include <functional>

// Boost includes
#include <boost/algorithm/string.hpp>
#include <boost/algorithm/string/split.hpp> // For manipulating paths

using namespace std;

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                          -- Constants --                                */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

#define THETA_LIMIT 1E-3

enum { LeftSide = 0, RightSide = 1, BottomSide = 2, TopSide = 3 };
enum { RightToLeftDiagonal = 0, LeftToRightDiagonal = 1 };

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                        -- Type declarations --                          */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

typedef double Array3x3[3][3];

class Line
{
 public:
  double EvaluateLineFunction(POS2D point);
  void SetEquation(POS2D *pt1, POS2D *pt2);
  void SetEquation(PixelCoords *pt1, PixelCoords *pt2);
  void GetEquation(double coefficients[3]);

  double a,b,c;
};

class Plane
{
 public:
  double EvaluatePlaneFunction(POS3D *point);
  void SetEquation(POS3D *pt1, POS3D *pt2, POS3D *pt3);
  void GetEquation(double coefficients[4]);
 private:
  double a,b,c,d;
};

class QuadtreeNode
{
 public:
  // - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
  QuadtreeNode(PixelCoords lowerLeft, PixelCoords upperRight);
  void SetTriangleGradients(BUFFER *buffer, int width);
  void SetTrianglePlanes(BUFFER *buffer, int width);
  void SetTriangles(int leftToRight, BUFFER *buffer, int width);
  void SetError(double error);
  void AverageDistanceError(BUFFER *buffer, int width, double *error);
  void MaxDistanceError(BUFFER *buffer, int width, double *error);
  void OldGradientError(BUFFER *buffer, int width, POS2D *diff);
  void AverageGradientError(BUFFER *buffer, int width, double *error);
  void MaxGradientError(BUFFER *buffer, int width, double *error);
  double CalculateError(BUFFER *buffer, int width);
  void PickMinErrorTriangles(BUFFER *buffer, int width);
  QuadtreeNode *GetChild(int verticalSide, int horizontalSide);
  void SetChildNeighbors();
  int Subdivide();
  int Descendants(list<QuadtreeNode *> *nodes);
  int Leaves(list<QuadtreeNode *> *leaves);
  bool IsLeaf();
  bool HasGrandChildren();
  bool IsAdjacent(QuadtreeNode *other, int *adjacentSide);
  QuadtreeNode *GetAncestralNeighbor(int side);
  //   bool operator<(QuadtreeNode &x) { return (m_error < x.m_error); }
  //   bool operator>(QuadtreeNode &x) { return (m_error > x.m_error); }
  //   bool operator==(QuadtreeNode &x) { return (m_error == x.m_error); }

  // - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
  QuadtreeNode *m_parent;
  list<QuadtreeNode *> *m_children;
  QuadtreeNode *m_neighbors[4];
  PixelCoords m_lowerLeft,m_upperRight;
  Line m_leftRightDiagonal,m_rightLeftDiagonal;
  bool m_diagonalIsLeftRight;
  POS2D m_triangleGradients[2];
  Plane m_trianglePlanes[2];
  bool m_goodSide[4];
  double m_error;
 protected:
 private:
  void SetExternalNeighbor(int parentSide, int childQualifier);
  bool OverLappingInterval(int aLeft, int aRight, int bLeft, int bRight);
};

class Quadtree
{
 public:
  Quadtree(BUFFER *buffer, int width, int height);
  void Subdivide(double threshold, int maxNumTriangles);
  void CreateTriangleMesh();
  QuadtreeNode *Root() { return m_root; }
  void Root(QuadtreeNode *root) { m_root = root; }

 protected:
 private:
  struct ContentsGreater
  {
    bool operator()(QuadtreeNode *x, QuadtreeNode *y)
    {
      return (x->m_error > y->m_error);
    }
  };
  QuadtreeNode *m_root;
  BUFFER *m_buffer;
  int m_bufferWidth;
  int m_bufferHeight;
};

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                 -- Forward function declarations --                     */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

static void AddQuadTriangles(QuadtreeNode *quad, BUFFER *buffer,
                             int width, int *numTriangles, int *numVertices,
                             int *indexMap);

static void MendTriangleMeshSeams(BUFFER *buffer, int width, int *indexMap,
                                  Quadtree *quadtree);

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*             -- Global variables private to this file --                 */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */


/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                      -- For Debugging --                                */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

// #if 0
// static void
// FindGradientMinMax(BUFFER *buffer, int size, double *min, double *max,
//                 char direction)
// {
//   pixel *disparities = buffer->disp;
//   int index;
//   double magGradient;

//   /* find first good pixel */
//   for (index = 0; index < size; index++)
//     if ((disparities[index] != 0) && (disparities[index] != MISSING_PIXEL) &&
//      (disparities[index] != FAR_FIELD_PIXEL))
//       break;

//   switch (direction)
//   {
//   case 'x':
//     *min = *max = buffer->gradients[index++].x;
//     for (; index < size; index++)
//     {
//       if ((disparities[index] != 0) && (disparities[index] != MISSING_PIXEL) &&
//        (disparities[index] != FAR_FIELD_PIXEL)) {
//      if (buffer->gradients[index].x < *min) {
//        *min = buffer->gradients[index].x;
//      }
//      else if (buffer->gradients[index].x > *max) {
//        *max = buffer->gradients[index].x;
//      }
//       }
//     }
//     break;
//   case 'y':
//     *min = *max = buffer->gradients[index++].y;
//     for (; index < size; index++)
//     {
//       if ((disparities[index] != 0) && (disparities[index] != MISSING_PIXEL) &&
//        (disparities[index] != FAR_FIELD_PIXEL)) {
//      if (buffer->gradients[index].y < *min) {
//           *min = buffer->gradients[index].y;
//         }
//         else if (buffer->gradients[index].y > *max) {
//        *max = buffer->gradients[index].y;
//      }
//       }
//     }
//     break;
//   case 'm':
//     *min = *max = sqrt(buffer->gradients[index].x*buffer->gradients[index].x +
//                     buffer->gradients[index].y*buffer->gradients[index].y);
//     index++;
//     for (; index < size; index++)
//     {
//       if ((disparities[index] != 0) && (disparities[index] != MISSING_PIXEL) &&
//        (disparities[index] != FAR_FIELD_PIXEL))
//       {
//      magGradient =
//        sqrt(buffer->gradients[index].x*buffer->gradients[index].x +
//             buffer->gradients[index].y*buffer->gradients[index].y);
//      if (magGradient < *min)
//        *min = magGradient;
//      else if (magGradient > *max)
//        *max = magGradient;
//       }
//     }
//   }
// }

// static void
// FindGradientMax(BUFFER *buffer, int size, double *max, char direction)
// {
//   pixel *disparities = buffer->disp;
//   int index;
//   double magGradient;

//   /* find first good pixel */
//   for (index = 0; index < size; index++)
//     if ((disparities[index] != 0) && (disparities[index] != MISSING_PIXEL) &&
//      (disparities[index] != FAR_FIELD_PIXEL))
//       break;

//   switch (direction)
//   {
//   case 'x':
//     *max = fabs(buffer->gradients[index++].x);
//     for (; index < size; index++)
//     {
//       if (buffer->gradients[index].x > *max)
//      *max = fabs(buffer->gradients[index].x);
//     }
//     break;
//   case 'y':
//     *max = fabs(buffer->gradients[index++].y);
//     for (; index < size; index++)
//     {
//       if ((disparities[index] != 0) && (disparities[index] != MISSING_PIXEL) &&
//        (disparities[index] != FAR_FIELD_PIXEL))
//      if (buffer->gradients[index].y > *max)
//        *max = fabs(buffer->gradients[index].y);
//     }
//     break;
//   case 'm':
//     *max = sqrt(buffer->gradients[index].x*buffer->gradients[index].x +
//              buffer->gradients[index].y*buffer->gradients[index].y);
//     index++;
//     for (; index < size; index++)
//     {
//       magGradient = sqrt(buffer->gradients[index].x*buffer->gradients[index].x +
//                       buffer->gradients[index].y*buffer->gradients[index].y);
//       if (magGradient > *max)
//      *max = magGradient;
//     }
//   }
// }

// static void
// MapGradientToImage(BUFFER *buffer, int size, double min, double max,
//                 int direction, pixel *imageBuffer)
// {
//   double absGrad, scale, offset, range = max - min;
//   pixel *disparities = buffer->disp;
//   double fudge = 0.5;
//   int i;

//   scale = 255.0/range;
//   offset = -min*scale;

//   switch (direction)
//   {
//   case 'x':
//     for (i=0; i < size; i++)
//     {
//       if ((disparities[i] != 0) && (disparities[i] != MISSING_PIXEL) &&
//        (disparities[i] != FAR_FIELD_PIXEL))
//       {
//      absGrad = fabs(buffer->gradients[i].x);
//      imageBuffer[i] = scale*absGrad + offset;
//       }
//       else
//      imageBuffer[i] = min;
//     }
//     break;
//   case 'y':
//     for (i=0; i < size; i++)
//     {
//       if ((disparities[i] != 0) && (disparities[i] != MISSING_PIXEL) &&
//        (disparities[i] != FAR_FIELD_PIXEL))
//       {
//      absGrad = fabs(buffer->gradients[i].y);
//      imageBuffer[i] = scale*absGrad + offset;
//       }
//       else
//      imageBuffer[i] = min;
//     }
//     break;
//   case 'm':
//     for (i=0; i < size; i++)
//     {
//       if ((disparities[i] != 0) && (disparities[i] != MISSING_PIXEL) &&
//        (disparities[i] != FAR_FIELD_PIXEL))
//       {
//      absGrad = sqrt(buffer->gradients[i].x*buffer->gradients[i].x +
//                     buffer->gradients[i].y*buffer->gradients[i].y);
//      if ((fudge*scale*absGrad + offset) <= 255.0)
//        imageBuffer[i] = fudge*scale*absGrad + offset;
//      else
//        imageBuffer[i] = 255;
//       }
//       else
//      imageBuffer[i] = min;
//     }
//     break;
//   }
// }
// #endif

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                        -- PixelCoord functions --                       */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

int
PixelCoords::PixelIndex(int imageWidth)
{
  return (row * imageWidth + col);
}

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                        -- Line functions --                             */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

double
Line::EvaluateLineFunction(POS2D point)
{
  /* If a^2 + b^2 == 1 then the value returned is the signed distance of */
  /* the point to the line. */
  /* If c < 0 then positive values mean the point is on the same side */
  /* as the normal. */
  return (a*point.x + b*point.y + c);
}

void
Line::SetEquation(POS2D *pt1, POS2D *pt2)
{
  double deltaX = pt2->x - pt1->x;
  double deltaY = pt2->y - pt1->y;

  a = deltaY;
  b = -deltaX;
  c = -a*pt1->x - b*pt1->y;
}

void
Line::SetEquation(PixelCoords *pt1, PixelCoords *pt2)
{
  double deltaX = pt2->col - pt1->col;
  double deltaY = pt2->row - pt1->row;

  a = deltaY;
  b = -deltaX;
  c = -a * ((double) pt1->col) - b * ((double) pt1->row);
}


/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                        -- Plane functions --                            */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

double
Plane::EvaluatePlaneFunction(POS3D *point)
{
  /* If a^2 + b^2 + c^2 == 1 then the value returned is the signed distance */
  /* of the point to the plane. */
  /* If d < 0 then positive values mean the point is on the same side */
  /* as the normal. */
  return (a*point->x + b*point->y + c*point->z + d);
}

void
Plane::SetEquation(POS3D *pt1, POS3D *pt2, POS3D *pt3)
{
  POS3D normal,*v1, *v2;
  double normalizer,mag;

  /* This is Newell's method for generating a plane equation */
  v1 = pt1;
  v2 = pt2;
  normal.x = (v1->y - v2->y) * (v1->z + v2->z);
  normal.y = (v1->z - v2->z) * (v1->x + v2->x);
  normal.z = (v1->x - v2->x) * (v1->y + v2->y);
  v1 = pt2;
  v2 = pt3;
  normal.x += (v1->y - v2->y) * (v1->z + v2->z);
  normal.y += (v1->z - v2->z) * (v1->x + v2->x);
  normal.z += (v1->x - v2->x) * (v1->y + v2->y);
  v1 = pt3;
  v2 = pt1;
  normal.x += (v1->y - v2->y) * (v1->z + v2->z);
  normal.y += (v1->z - v2->z) * (v1->x + v2->x);
  normal.z += (v1->x - v2->x) * (v1->y + v2->y);

  if ((mag = sqrt(normal.x*normal.x + normal.y*normal.y +
                  normal.z*normal.z)) == 0)
  {
    /* If this happens points are geometrically coincident, and we
       arbitrarily pick a normal */
    /* printf("Warning in SetEquation: points are "
       "geometrically concident.\n"); */
    a = b = 0.0;
    c = 1.0;
    d = 0.0;
    return;
  }
  normalizer = 1.0/mag;
  a = normal.x * normalizer;
  b = normal.y * normalizer;
  c = normal.z * normalizer;
  d = -(pt1->x*a + pt1->y*b + pt1->z*c);
}

void
Plane::GetEquation(double coefficients[4])
{
  coefficients[0] = a;
  coefficients[1] = b;
  coefficients[2] = c;
  coefficients[3] = d;
}

/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                     -- QuadtreeNode Functions --                        */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

QuadtreeNode::QuadtreeNode(PixelCoords lowerLeft, PixelCoords upperRight)
{                               /* QuadtreeNode constructor */
  PixelCoords lowerRight(upperRight.col,lowerLeft.row);
  PixelCoords upperLeft(lowerLeft.col,upperRight.row);

  m_parent = 0;
  m_children = new list<QuadtreeNode *>;
  m_neighbors[0] = m_neighbors[1] = m_neighbors[2] = m_neighbors[3] = 0;

  m_lowerLeft = lowerLeft;
  m_upperRight = upperRight;

  /* diagonal from lower left to upper right */
  m_leftRightDiagonal.SetEquation(&lowerLeft,&upperRight);
  /* diagonal from lower right to upper left */
  m_rightLeftDiagonal.SetEquation(&lowerRight,&upperLeft);

  /* arbitrarily choose one diagonal initially */
  m_diagonalIsLeftRight = 1;
  m_goodSide[0] = m_goodSide[1] = m_goodSide[2] = m_goodSide[3] = false;
  m_error = 0.0;
}

void
QuadtreeNode::SetTriangleGradients(BUFFER *buffer, int width)
{
  POS3D lowerLeft,lowerRight,upperLeft,upperRight;
  int pixelIndex, rowStartIndex = m_lowerLeft.row * width;

  pixelIndex = rowStartIndex + m_lowerLeft.col;
  lowerLeft = buffer->dot[pixelIndex];
  pixelIndex = rowStartIndex + m_upperRight.col;
  lowerRight = buffer->dot[pixelIndex];

  rowStartIndex = m_upperRight.row *width;
  pixelIndex = rowStartIndex + m_lowerLeft.col;
  upperLeft = buffer->dot[pixelIndex];
  pixelIndex = rowStartIndex + m_upperRight.col;
  upperRight = buffer->dot[pixelIndex];

  if (m_diagonalIsLeftRight)
  {
    m_triangleGradients[0].x = lowerRight.z - lowerLeft.z;
    m_triangleGradients[0].y = upperRight.z - lowerRight.z;

    m_triangleGradients[1].x = upperRight.z - upperLeft.z;
    m_triangleGradients[1].y = upperLeft.z - lowerLeft.z;
  }
  else
  {
    m_triangleGradients[0].x = upperRight.z - upperLeft.z;
    m_triangleGradients[0].y = upperRight.z - lowerRight.z;

    m_triangleGradients[1].x = lowerRight.z - lowerLeft.z;
    m_triangleGradients[1].y = upperLeft.z - lowerLeft.z;
  }
}

void
QuadtreeNode::SetTrianglePlanes(BUFFER *buffer, int width)
{
  POS3D lowerLeft,lowerRight,upperLeft,upperRight;
  int pixelIndex, rowStartIndex = m_lowerLeft.row * width;

  pixelIndex = rowStartIndex + m_lowerLeft.col;
  lowerLeft = buffer->dot[pixelIndex];
  pixelIndex = rowStartIndex + m_upperRight.col;
  lowerRight = buffer->dot[pixelIndex];

  rowStartIndex = m_upperRight.row *width;
  pixelIndex = rowStartIndex + m_lowerLeft.col;
  upperLeft = buffer->dot[pixelIndex];
  pixelIndex = rowStartIndex + m_upperRight.col;
  upperRight = buffer->dot[pixelIndex];

  if (m_diagonalIsLeftRight)
  {
    m_trianglePlanes[0].SetEquation(&lowerLeft,&lowerRight,&upperRight);
    m_trianglePlanes[1].SetEquation(&upperLeft,&lowerLeft,&upperRight);
  }
  else
  {
    m_trianglePlanes[0].SetEquation(&upperRight,&upperLeft,&lowerRight);
    m_trianglePlanes[1].SetEquation(&lowerLeft,&lowerRight,&upperLeft);
  }
}

void
QuadtreeNode::SetTriangles(int leftToRight, BUFFER *buffer, int width)
{
  m_diagonalIsLeftRight = leftToRight;
  SetTriangleGradients(buffer,width);
  SetTrianglePlanes(buffer,width);
}

void
QuadtreeNode::SetError(double error)
{
  m_error = error;
}

void
QuadtreeNode::AverageDistanceError(BUFFER *buffer, int width, double *error)
{
  int row,col,rowStartIndex,pixelIndex,numPixels;
  int minRow,minCol,stopRow,stopCol,quadHeight,quadWidth;
  POS2D point;
  Line *diagonal;
  double d;

  quadWidth = m_upperRight.col - m_lowerLeft.col + 1;
  quadHeight = m_upperRight.row - m_lowerLeft.row + 1;
  numPixels = quadWidth*quadHeight;
  minRow = m_lowerLeft.row;
  minCol = m_lowerLeft.col;
  stopRow = minRow+quadHeight;
  stopCol = minCol+quadWidth;
  rowStartIndex = minRow*width;

  diagonal = (m_diagonalIsLeftRight ?
              &(m_leftRightDiagonal) : &(m_rightLeftDiagonal));

  *error = 0.0;
  for (row = minRow; row < stopRow; rowStartIndex += width, row++)
  {
    for (col = minCol; col < stopCol; col++)
    {
      pixelIndex = rowStartIndex+col;
      point.x = col;
      point.y = row;

      if ((d = diagonal->EvaluateLineFunction(point)) > 0.0)
        *error += fabs(m_trianglePlanes[0].
                       EvaluatePlaneFunction(&buffer->dot[pixelIndex]));
      else if (d <= 0.0)
        *error += fabs(m_trianglePlanes[1].
                       EvaluatePlaneFunction(&buffer->dot[pixelIndex]));
    }
  }
  /* we average the distance error... */
  *error = *error / (double) numPixels;
}

void
QuadtreeNode::MaxDistanceError(BUFFER *buffer, int width, double *error)
{
  int row,col,rowStartIndex,pixelIndex;
  int minRow,minCol,stopRow,stopCol,quadHeight,quadWidth;
  POS2D point;
  Line *diagonal;
  double dist;

  quadWidth = m_upperRight.col - m_lowerLeft.col + 1;
  quadHeight = m_upperRight.row - m_lowerLeft.row + 1;
  minRow = m_lowerLeft.row;
  minCol = m_lowerLeft.col;
  stopRow = minRow+quadHeight;
  stopCol = minCol+quadWidth;
  rowStartIndex = minRow*width;

  diagonal = (m_diagonalIsLeftRight ?
              &(m_leftRightDiagonal) : &(m_rightLeftDiagonal));

  *error = 0.0;
  for (row = minRow; row < stopRow; rowStartIndex += width, row++)
  {
    for (col = minCol; col < stopCol; col++)
    {
      pixelIndex = rowStartIndex+col;
      point.x = col;
      point.y = row;

      if ((diagonal->EvaluateLineFunction(point)) > 0.0)
        dist = fabs(m_trianglePlanes[0].
                    EvaluatePlaneFunction(&buffer->dot[pixelIndex]));
      else
        dist = fabs(m_trianglePlanes[1].
                    EvaluatePlaneFunction(&buffer->dot[pixelIndex]));
      if (dist > *error)
        *error = dist;
    }
  }
}

/* Can do this calc. (at least) two different ways: find the max */
/* error, or find the average error. I think the latter method */
/* should give better results */

void
QuadtreeNode::OldGradientError(BUFFER *buffer, int width, POS2D *diff)
{
  int row,col,rowStartIndex,pixelIndex,numPixels;
  int minRow,minCol,stopRow,stopCol,quadHeight,quadWidth;
  POS2D point;
  Line *diagonal;
  double d;

  quadWidth = m_upperRight.col - m_lowerLeft.col + 1;
  quadHeight = m_upperRight.row - m_lowerLeft.row + 1;
  numPixels = quadWidth*quadHeight;
  minRow = m_lowerLeft.row;
  minCol = m_lowerLeft.col;
  stopRow = minRow+quadHeight;
  stopCol = minCol+quadWidth;
  rowStartIndex = minRow*width;

  diagonal = (m_diagonalIsLeftRight ?
              &(m_leftRightDiagonal) : &(m_rightLeftDiagonal));

  diff->x = diff->y = 0.0;
  for (row = minRow; row < stopRow; rowStartIndex += width, row++)
  {
    for (col = minCol; col < stopCol; col++)
    {
      pixelIndex = rowStartIndex+col;
      point.x = col;
      point.y = row;

      if ((d = diagonal->EvaluateLineFunction(point)) > 0.0)
      {
        diff->x += buffer->gradients[pixelIndex].x - m_triangleGradients[0].x;
        diff->y += buffer->gradients[pixelIndex].y - m_triangleGradients[0].y;
      }
      else if (d < 0.0)
      {
        diff->x += buffer->gradients[pixelIndex].x - m_triangleGradients[1].x;
        diff->y += buffer->gradients[pixelIndex].y - m_triangleGradients[1].y;
      }
      else                      /* we average the triangle gradients */
      {
        diff->x += (buffer->gradients[pixelIndex].x -
                    0.5*(m_triangleGradients[0].x + m_triangleGradients[1].x));

        diff->y += (buffer->gradients[pixelIndex].y -
                    0.5*(m_triangleGradients[0].y + m_triangleGradients[1].y));
      }
    }
  }
  /* we average the gradient error... */
  diff->x = diff->x / (double) numPixels;
  diff->y = diff->y / (double) numPixels;
}

void
QuadtreeNode::AverageGradientError(BUFFER *buffer, int width, double *error)
{
  int row,col,rowStartIndex,pixelIndex,numPixels;
  int minRow,minCol,stopRow,stopCol,quadHeight,quadWidth;
  POS2D diff,point;
  Line *diagonal;
  double d;

  quadWidth = m_upperRight.col - m_lowerLeft.col + 1;
  quadHeight = m_upperRight.row - m_lowerLeft.row + 1;
  numPixels = quadWidth*quadHeight;
  minRow = m_lowerLeft.row;
  minCol = m_lowerLeft.col;
  stopRow = minRow+quadHeight;
  stopCol = minCol+quadWidth;
  rowStartIndex = minRow*width;

  diagonal = (m_diagonalIsLeftRight ?
              &(m_leftRightDiagonal) : &(m_rightLeftDiagonal));

  *error = 0.0;
  for (row = minRow; row < stopRow; rowStartIndex += width, row++)
  {
    for (col = minCol; col < stopCol; col++)
    {
      pixelIndex = rowStartIndex+col;
      point.x = col;
      point.y = row;

      if ((d = diagonal->EvaluateLineFunction(point)) > 0.0)
      {
        diff.x = buffer->gradients[pixelIndex].x - m_triangleGradients[0].x;
        diff.y = buffer->gradients[pixelIndex].y - m_triangleGradients[0].y;
      }
      else if (d < 0.0)
        {
          diff.x = buffer->gradients[pixelIndex].x - m_triangleGradients[1].x;
          diff.y = buffer->gradients[pixelIndex].y - m_triangleGradients[1].y;
        }
      else                      /* we average the triangle gradients */
        {
          diff.x = (buffer->gradients[pixelIndex].x -
                    0.5*(m_triangleGradients[0].x + m_triangleGradients[1].x));

          diff.y = (buffer->gradients[pixelIndex].y -
                    0.5*(m_triangleGradients[0].y + m_triangleGradients[1].y));
        }
      *error += sqrt(diff.x*diff.x + diff.y*diff.y);
    }
  }
  /* we average the gradient error... */
  *error = *error / (double) numPixels;
}

void
QuadtreeNode::MaxGradientError(BUFFER *buffer, int width, double *error)
{
  int row,col,rowStartIndex,pixelIndex;
  int minRow,minCol,stopRow,stopCol,quadHeight,quadWidth;
  POS2D diff,point;
  Line *diagonal;
  double d,magSqrDiff;

  quadWidth = m_upperRight.col - m_lowerLeft.col + 1;
  quadHeight = m_upperRight.row - m_lowerLeft.row + 1;
  minRow = m_lowerLeft.row;
  minCol = m_lowerLeft.col;
  stopRow = minRow+quadHeight;
  stopCol = minCol+quadWidth;
  rowStartIndex = minRow*width;

  diagonal = (m_diagonalIsLeftRight ?
              &(m_leftRightDiagonal) : &(m_rightLeftDiagonal));

  *error = 0.0;
  for (row = minRow; row < stopRow; rowStartIndex += width, row++)
    {
      for (col = minCol; col < stopCol; col++)
    {
      pixelIndex = rowStartIndex+col;
      point.x = col;
      point.y = row;

      if ((d = diagonal->EvaluateLineFunction(point)) > 0.0)
      {
        diff.x = buffer->gradients[pixelIndex].x - m_triangleGradients[0].x;
        diff.y = buffer->gradients[pixelIndex].y - m_triangleGradients[0].y;
      }
      else if (d < 0.0)
      {
        diff.x = buffer->gradients[pixelIndex].x - m_triangleGradients[1].x;
        diff.y = buffer->gradients[pixelIndex].y - m_triangleGradients[1].y;
      }
      else                      /* we average the triangle gradients */
      {
        diff.x = (buffer->gradients[pixelIndex].x -
                  0.5*(m_triangleGradients[0].x + m_triangleGradients[1].x));

        diff.y = (buffer->gradients[pixelIndex].y -
                  0.5*(m_triangleGradients[0].y + m_triangleGradients[1].y));
      }
      if ((magSqrDiff = (diff.x*diff.x + diff.y*diff.y)) > *error)
        *error = magSqrDiff;
    }
  }
  *error = sqrt(*error);
}

double
QuadtreeNode::CalculateError(BUFFER *buffer, int width)
{
  double error;

  AverageDistanceError(buffer,width,&error);
#if 0
  MaxDistanceError(buffer,width,&error);
  MaxGradientError(buffer,width,&error);
  AverageGradientError(buffer,width,&error);
#endif
  return (error);
}

void
QuadtreeNode::PickMinErrorTriangles(BUFFER *buffer, int width)
{
  double error,minError;

  SetTriangles(RightToLeftDiagonal,buffer,width);
  minError = CalculateError(buffer,width);

  SetTriangles(LeftToRightDiagonal,buffer,width);
  if ((error = CalculateError(buffer,width)) < minError)
    minError = error;
  else                          /* use the original 2 triangles we tried */
    SetTriangles(RightToLeftDiagonal,buffer,width);

  SetError(minError);
}

QuadtreeNode *
QuadtreeNode::GetChild(int verticalSide, int horizontalSide)
{
  int numChildren,quadWidth,i,iStop;
  list<QuadtreeNode *>::iterator element;

  if (m_children->empty())
    return (0);

  numChildren = m_children->size();
  element = m_children->begin();
  if (numChildren == 4)
  {
    if (horizontalSide == LeftSide)
      iStop = (verticalSide == TopSide) ? 2 : 0;
    else
      iStop = (verticalSide == TopSide) ? 3 : 1;
  }
  else if (numChildren == 2)
  {
    quadWidth = m_upperRight.col - m_lowerLeft.col + 1;
    if (quadWidth > 2)
      iStop = (horizontalSide == LeftSide) ? 0 : 1;
    else
      iStop = (verticalSide == BottomSide) ? 0 : 1;
  }
  else                          /* should never happen */
  {
    printf("ERROR in GetChild: numChildren(=%d) is not 0, 2, or 4. "
           "Aborting.\n",numChildren);
    return (0);
  }

  for (i = 0; i < iStop; i++)
    ++element;

  return (*element);
}

void
QuadtreeNode::SetExternalNeighbor(int parentSide, int childQualifier)
{
  QuadtreeNode *neighborChild;
  int neighborSide;

  if (m_parent == 0)
    return;

  if (m_parent->m_neighbors[parentSide] == 0)
  {
    m_neighbors[parentSide] = 0;
    return;
  }
  if (parentSide > RightSide)
  {
    neighborSide = BottomSide+TopSide - parentSide;
    neighborChild =
      m_parent->m_neighbors[parentSide]->GetChild(neighborSide,childQualifier);
  }
  else
  {
    neighborSide = LeftSide+RightSide - parentSide;
    neighborChild =
      m_parent->m_neighbors[parentSide]->GetChild(childQualifier,neighborSide);
  }

  m_neighbors[parentSide] = neighborChild;
  if (neighborChild != 0)
    neighborChild->m_neighbors[neighborSide] = this;
}

void
QuadtreeNode::SetChildNeighbors()
{
  QuadtreeNode *children[4];
  list<QuadtreeNode *>::iterator element;
  int i,numChildren, quadWidth = m_upperRight.col - m_lowerLeft.col + 1;

  for (i = 0, element = m_children->begin(); !(element == m_children->end());
       element++, i++)
    children[i] = *element;
  numChildren = i;

  /* external neighbors common to both 4 child and 2 child subdivisions */
  children[0]->SetExternalNeighbor(LeftSide,BottomSide);
  children[0]->SetExternalNeighbor(BottomSide,LeftSide);
  if (numChildren == 4)
  {
    /* external neighbors */
    children[1]->SetExternalNeighbor(BottomSide,RightSide);
    children[1]->SetExternalNeighbor(RightSide,BottomSide);
    children[2]->SetExternalNeighbor(LeftSide,TopSide);
    children[2]->SetExternalNeighbor(TopSide,LeftSide);
    children[3]->SetExternalNeighbor(RightSide,TopSide);
    children[3]->SetExternalNeighbor(TopSide,RightSide);
    /* internal neigbors */
    children[0]->m_neighbors[RightSide] = children[1];
    children[1]->m_neighbors[LeftSide] = children[0];
    children[0]->m_neighbors[TopSide] = children[2];
    children[2]->m_neighbors[BottomSide] = children[0];
    children[1]->m_neighbors[TopSide] = children[3];
    children[3]->m_neighbors[BottomSide] = children[1];
    children[2]->m_neighbors[RightSide] = children[3];
    children[3]->m_neighbors[LeftSide] = children[2];
  }
  else if (numChildren == 2)    /* numChildren should always be 2 or 4 */
  {
    if (quadWidth < 3)
    {
      /* external neighbors */
      children[0]->SetExternalNeighbor(RightSide,BottomSide);
      children[1]->SetExternalNeighbor(LeftSide,TopSide);
      children[1]->SetExternalNeighbor(RightSide,TopSide);
      children[1]->SetExternalNeighbor(TopSide,RightSide);
      /* internal neighbors */
      children[1]->m_neighbors[BottomSide] = children[0];
      children[0]->m_neighbors[TopSide] = children[1];
    }
    else
    {
      /* external neighbors */
      children[0]->SetExternalNeighbor(TopSide,LeftSide);
      children[1]->SetExternalNeighbor(RightSide,TopSide);
      children[1]->SetExternalNeighbor(TopSide,RightSide);
      children[1]->SetExternalNeighbor(BottomSide,RightSide);
      children[1]->m_neighbors[RightSide] = m_neighbors[RightSide];
      children[1]->m_neighbors[TopSide] = m_neighbors[TopSide];
      children[1]->m_neighbors[BottomSide] = m_neighbors[BottomSide];

      /* internal neighbors */
      children[0]->m_neighbors[RightSide] = children[1];
      children[1]->m_neighbors[LeftSide] = children[0];
    }
  }
  else
  {
    printf("ERROR in SetChildNeighbors: numChildren is not 2 or 4. "
           "Aborting.\n");
  }
}

int
QuadtreeNode::Subdivide()
{
  PixelCoords leftMiddle,lowerMiddle,rightMiddle,upperMiddle,middle;
  QuadtreeNode *newQuadtreeNodes[4] = {0,0,0,0};
  int quadWidth,quadHeight, numChildren = 0;
//   int i,j;                   /* debugging */

  if (!m_children->empty()) /* only subdivide leaf nodes */
  {
    printf("WARNING in Subdivide(): this QuadtreeNode already has children. "
           "Aborting.\n");
    return (numChildren);
  }

  /* cannot subdivide if quad has <= 4 pixels */
  /* also must have at least 2 pixels in both dimensions */
  quadWidth = m_upperRight.col - m_lowerLeft.col + 1;
  quadHeight = m_upperRight.row - m_lowerLeft.row + 1;
  if ((quadWidth < 2) || (quadHeight < 2) ||
      ((quadWidth < 3) && (quadHeight < 3)))
    return (numChildren);

  /* set indices of new quad corners */
  middle.row = (m_lowerLeft.row + m_upperRight.row)/2;
  middle.col = (m_lowerLeft.col + m_upperRight.col)/2;
  if (quadHeight == 2)
    middle.row += 1;
  if (quadWidth == 2)
    middle.col += 1;
  leftMiddle.row = middle.row;
  leftMiddle.col = m_lowerLeft.col;
  rightMiddle.row = middle.row;
  rightMiddle.col = m_upperRight.col;
  lowerMiddle.row = m_lowerLeft.row;
  lowerMiddle.col = middle.col;
  upperMiddle.row = m_upperRight.row;
  upperMiddle.col = middle.col;

  /* the order of the children will be:
     lower-left, lower-right, upper-left, upper-right */

  /*
   * If the mesh elements are much wider than they are long, or vice versa,
   * we should subdivide the mesh in the direction that makes the elements more
   * square.
   */
  if (quadWidth > (2 * quadHeight))
  {
    newQuadtreeNodes[0] = new QuadtreeNode(m_lowerLeft,upperMiddle);
    newQuadtreeNodes[0]->m_parent = this;
    m_children->push_back(newQuadtreeNodes[0]);
    numChildren++;

    if (quadWidth > 2)
    {
      newQuadtreeNodes[1] = new QuadtreeNode(lowerMiddle,m_upperRight);
      newQuadtreeNodes[1]->m_parent = this;
      m_children->push_back(newQuadtreeNodes[1]);
      numChildren++;
    }
  }
  else if (quadHeight > (2 * quadWidth))
  {
    newQuadtreeNodes[0] = new QuadtreeNode(m_lowerLeft, rightMiddle);
    newQuadtreeNodes[0]->m_parent = this;
    m_children->push_back(newQuadtreeNodes[0]);
    numChildren++;

    if (quadHeight > 2)
    {
      newQuadtreeNodes[1] = new QuadtreeNode(leftMiddle,m_upperRight);
      newQuadtreeNodes[1]->m_parent = this;
      m_children->push_back(newQuadtreeNodes[1]);
      numChildren++;
    }
  }
  else  /* The aspect ratio of the quads is fairly square */
  {
    newQuadtreeNodes[0] = new QuadtreeNode(m_lowerLeft,middle);
    newQuadtreeNodes[0]->m_parent = this;
    m_children->push_back(newQuadtreeNodes[0]);
    numChildren++;

    if (quadWidth > 2)
    {
      newQuadtreeNodes[1] = new QuadtreeNode(lowerMiddle,rightMiddle);
      newQuadtreeNodes[1]->m_parent = this;
      m_children->push_back(newQuadtreeNodes[1]);
      numChildren++;
    }

    if (quadHeight > 2)
    {
      newQuadtreeNodes[numChildren] = new QuadtreeNode(leftMiddle,upperMiddle);
      newQuadtreeNodes[numChildren]->m_parent = this;
      m_children->push_back(newQuadtreeNodes[numChildren]);
      numChildren++;
    }

    if ((quadWidth > 2) && (quadHeight > 2))
    {
      newQuadtreeNodes[3] = new QuadtreeNode(middle,m_upperRight);
      newQuadtreeNodes[3]->m_parent = this;
      m_children->push_back(newQuadtreeNodes[3]);
      numChildren++;
    }
  }

  SetChildNeighbors();

//   for (i = 0; i < numChildren; i++)
//     for (j = 0; j < numChildren; j++)
//       if ((newQuadtreeNodes[i] != 0) &&
//        (newQuadtreeNodes[i]->m_neighbors[j] != 0))
//      assert(IsAdjacent(newQuadtreeNodes[i],
//                        newQuadtreeNodes[i]->m_neighbors[j],&j));

  return (numChildren);
}

/* FIX: Larry should really document what funky recursive code like
   this is supposed to do. */

/* Descendants finds all the nodes below a given node in a Quadtree
   and places them in a list.
   The tree is traversed in preorder fashion.
*/
int
QuadtreeNode::Descendants(list<QuadtreeNode *> *nodes)
{
  static int recurseLevel = 0;
  static int numNodes = 0;
  QuadtreeNode *childNode;
  list<QuadtreeNode *>::iterator element,endElement;
  int returnValue;

  ++recurseLevel;
  ++numNodes;

  nodes->push_back(this);

  element = m_children->begin();
  endElement = m_children->end();
  while (!(element == endElement))
  {
    childNode = *element;
    childNode->Descendants(nodes);
    ++element;
  }
  --recurseLevel;

  returnValue = numNodes;
  if (0 == recurseLevel)
    numNodes = 0;

  return (returnValue);
}

int
QuadtreeNode::Leaves(list<QuadtreeNode *> *leaves)
{
  list<QuadtreeNode *>::iterator element,endElement,nextElement;
  QuadtreeNode *node;
  unsigned int numLeaves;

  numLeaves = Descendants(leaves);
  assert(leaves->size() == numLeaves);

  element = leaves->begin();
  endElement = leaves->end();
  while (!(element == endElement))
  {
    nextElement = element;
    ++nextElement;

    node = *element;

    if (!node->m_children->empty())
    {
      leaves->erase(element);
      --numLeaves;

      if (0 > numLeaves)
      {
        fprintf (stdout,"Leaves: numLeaves < 0 (%d)\n",numLeaves);
        abort();
      }
    }

    element = nextElement;
  }
  return (numLeaves);
}

bool
QuadtreeNode::IsLeaf()
{
  return (m_children->empty());
}

bool
QuadtreeNode::HasGrandChildren()
{
  list<QuadtreeNode *>::iterator element;
  QuadtreeNode *childQuadtreeNode;

  element = m_children->begin();
  while (!(element == m_children->end()))
  {
    childQuadtreeNode = *element;
    if (!childQuadtreeNode->IsLeaf())
      return (true);
    ++element;
  }
  return (false);
}

QuadtreeNode *
QuadtreeNode::GetAncestralNeighbor(int side)
{
  QuadtreeNode *neighbor;

  if (m_parent == 0)
    return (0);

  if ((neighbor = m_parent->m_neighbors[side]) != 0)
    return (neighbor);
  else
    return (m_parent->GetAncestralNeighbor(side));
}

bool
QuadtreeNode::OverLappingInterval(int aLeft, int aRight, int bLeft, int bRight)
{
  return (((aRight <= bRight) && (aLeft >= bLeft)) ||
          ((aRight >= bRight) && (aLeft <= bLeft)));
}

bool
QuadtreeNode::IsAdjacent(QuadtreeNode *other, int *adjacentSide)
{
  /* if side is passed (i.e. != -1) we only look for adjacency on that side */
  if (*adjacentSide == -1)
  {
    if ((m_lowerLeft.row == other->m_upperRight.row) &&
        OverLappingInterval(m_lowerLeft.col,m_upperRight.col,
                            other->m_lowerLeft.col,
                            other->m_upperRight.col))
      *adjacentSide = BottomSide;
    else if ((m_upperRight.row == other->m_lowerLeft.row) &&
             OverLappingInterval(m_lowerLeft.col,m_upperRight.col,
                                 other->m_lowerLeft.col,
                                 other->m_upperRight.col))
      *adjacentSide = TopSide;

    else if ((m_upperRight.col == other->m_lowerLeft.col) &&
             OverLappingInterval(m_lowerLeft.row,m_upperRight.row,
                                 other->m_lowerLeft.row,
                                 other->m_upperRight.row))
      *adjacentSide = RightSide;
    else if ((m_lowerLeft.col == other->m_upperRight.col) &&
             OverLappingInterval(m_lowerLeft.row,m_upperRight.row,
                                 other->m_lowerLeft.row,
                                 other->m_upperRight.row))
      *adjacentSide = LeftSide;
    else
      *adjacentSide = -1;
  }
  else
  {
    switch (*adjacentSide)
    {
    case BottomSide:
      if ((m_lowerLeft.row != other->m_upperRight.row) ||
          !OverLappingInterval(m_lowerLeft.col,m_upperRight.col,
                               other->m_lowerLeft.col,other->m_upperRight.col))
        *adjacentSide = -1;
      break;
    case TopSide:
      if ((m_upperRight.row != other->m_lowerLeft.row) ||
          !OverLappingInterval(m_lowerLeft.col,m_upperRight.col,
                               other->m_lowerLeft.col,other->m_upperRight.col))
        *adjacentSide = -1;
      break;
    case RightSide:
      if ((m_upperRight.col != other->m_lowerLeft.col) ||
          !OverLappingInterval(m_lowerLeft.row,m_upperRight.row,
                               other->m_lowerLeft.row,other->m_upperRight.row))
        *adjacentSide = -1;
      break;
    case LeftSide:
      if ((m_lowerLeft.col != other->m_upperRight.col) ||
          !OverLappingInterval(m_lowerLeft.row,m_upperRight.row,
                               other->m_lowerLeft.row,other->m_upperRight.row))
        *adjacentSide = -1;
      break;
    default:
      *adjacentSide = -1;
      break;
    }
  }
  return (*adjacentSide != -1);
}


/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */
/*                       -- Quadtree Functions --                          */
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - */

/* Quadtree constructor */
Quadtree::Quadtree(BUFFER *buffer, int width, int height)
{
  PixelCoords lowerLeft,upperRight;

  lowerLeft.row = 0;
  lowerLeft.col = 0;
  upperRight.row = height - 1;
  upperRight.col = width - 1;

  m_buffer = buffer;
  m_bufferWidth = width;
  m_bufferHeight = height;

  m_root = new QuadtreeNode(lowerLeft,upperRight);

  m_root->m_parent = 0;
  /* pick the diagonal of the root that minimizes error */
  m_root->PickMinErrorTriangles(buffer,width);
}

#if 0
/* depth-first Recursive version */
void
Quadtree::Subdivide(QuadtreeNode *quad, double threshold)
{
  list<QuadtreeNode *>::iterator element;
  QuadtreeNode *childQuadtreeNode;

  if (m_error < threshold)
    return;
  else
  {
    quad->Subdivide();          // Subdivide this quadtreenode
    element = quad->m_children->begin();
    while (!(element == quad->m_children->end()))
    {
      childQuadtreeNode = *element;
      childQuadtreeNode->PickMinErrorTriangles(m_buffer,m_bufferWidth);
      Subdivide(childQuadtreeNode,threshold);
      ++element;
    }
  }
}
#endif

/* breadth-first iterative version */


/*
 * NOTE :
 *
 * Eventually we plan to switch over from using an STL list to an STL multiset
 * for storing the qElements.  There is some commented out code in here to
 * that effect, however it does not yet work properly.  At some point in the future
 * we'll take another look.  -Larry and Michael
 */
void
Quadtree::Subdivide(double threshold, int maxNumTriangles)
{
  /*   For gradient: */
  /*   static const double threshold = 0.0077; */
  /*   static const int maxTriangles = 500000; */
  /*   static const double threshold = -0.5; */
  /*   static const int maxTriangles = 40; */
  /*   static const int maxTriangles = 500000; */

  list<QuadtreeNode *>::iterator qElement, element, endElement;
//   list<QuadtreeNode *>::iterator element, endElement;
//   multiset<QuadtreeNode *>::iterator qElement;
  QuadtreeNode *childQuadtreeNode,*quadtreeNode;
  list<QuadtreeNode *> *queue = new list<QuadtreeNode *>;
//   multiset<QuadtreeNode *,ContentsGreater> *queue = new multiset<QuadtreeNode *,ContentsGreater>;
  int numTriangles = 0, maxQueueSize = 0, queueSize = 0;    /* debugging */
  int numChildren;

  queue->push_back(m_root);
//   queue->insert(m_root);
  queueSize = maxQueueSize = 1;

  numTriangles += 1*2;
  while (!queue->empty() && (numTriangles < maxNumTriangles))
  {
    qElement = queue->begin();
    quadtreeNode = *qElement;
    queue->pop_front();
//     queue->erase(queue->begin());
    --queueSize;

    if (quadtreeNode->m_error > threshold)
    {
      if ((numChildren = quadtreeNode->Subdivide()) > 0)
      {
        numTriangles -= (1*2);
        numTriangles += (numChildren*2);
      }
      element = quadtreeNode->m_children->begin();
      endElement = quadtreeNode->m_children->end();
      while (!(element == endElement))
      {
        childQuadtreeNode = *element;
        childQuadtreeNode->PickMinErrorTriangles(m_buffer,m_bufferWidth);
        queue->push_back(childQuadtreeNode);
//      queue->insert(childQuadtreeNode);
        ++queueSize;
        element = ++element;
      }
    }
    if (queueSize > maxQueueSize)
      maxQueueSize = queueSize;
  }
  queue->clear();
  delete queue;

  printf("Max queue size = %d\n",maxQueueSize);
  printf("Number of triangles = %d\n",numTriangles);
}

void
Quadtree::CreateTriangleMesh()
{
  list<QuadtreeNode *> *leaves = new list<QuadtreeNode *>;
  list<QuadtreeNode *>::iterator element,endElement;
  int numLeaves = 0, numVertices = 0, numPoints = m_bufferWidth*m_bufferHeight;
  int *indexMap,pixelIndex,i,j,row,col,rowStart, numTriangles = 0;
  float deltaU,deltaV,v;
  QuadtreeNode *leaf;

  numLeaves = m_root->Leaves(leaves);

  if (0 > numLeaves)
  {
    fprintf(stderr,"CreateTriangleMesh: numLeave=%d\n",numLeaves);
    abort();
  }

  /* Allocate memory for index map */
  if ((indexMap = (int *) malloc(numPoints*sizeof(int))) == NULL)
  {
    fprintf(stderr, "triangle mesh index map allocation failed.\n");
    exit(EXIT_FAILURE);
  }
  for (i = 0; i < numPoints; i++) /* init so we can tell unused indices */
    indexMap[i] = -1;

  /* Allocate memory for triangle buffer */
  if ((m_buffer->nff.triangle =
       (NFF_TR *) malloc(2*numLeaves*sizeof(NFF_TR))) == NULL)
  {
    fprintf(stderr, "CreateTriangleMesh: malloc numLeaves=%d\n",numLeaves );
    fprintf(stderr, "nff triangle buffer allocation failed.\n");
    exit(EXIT_FAILURE);
  }
  /* Generate triangle list */
  element = leaves->begin();
  endElement = leaves->end();
  while (!(element == endElement))
  {
    leaf = *element;
    AddQuadTriangles(leaf,m_buffer,m_bufferWidth,
                     &numTriangles,&numVertices,indexMap);
    ++element;
  }

  /* Allocate memory for vertex buffer */
  m_buffer->nff.pt_number = numVertices;
  if ((m_buffer->nff.vtx =
       (POS3D *) malloc(numVertices * sizeof(POS3D))) == NULL)
  {
    fprintf(stderr, "nff vertex buffer allocation failed.\n");
    exit(EXIT_FAILURE);
  }
  /* Set vertex coordinates */
  for (i = 0, j = 0; i < numPoints; i++)
  {
    if (indexMap[i] != -1)
    {
      m_buffer->nff.vtx[indexMap[i]] = m_buffer->dot[i];
      if (++j >= numVertices)
        break;
    }
  }

  MendTriangleMeshSeams(m_buffer,m_bufferWidth,indexMap,this);

  /* Allocate memory for texture coord. buffer */
  if ((m_buffer->nff.tex =
       (UV_TEX *) malloc(numVertices * sizeof(UV_TEX))) == NULL)
  {
    fprintf(stderr, "nff texture buffer allocation failed.\n");
    exit(EXIT_FAILURE);
  }
  /* Generate texture coordinates */
  deltaU = 1.0/(float) m_bufferWidth;
  deltaV = 1.0/(float) m_bufferHeight;
  for (row = 0, rowStart = 0, j = 0; row < m_bufferHeight; row++)
  {
    v = deltaV * (float) (m_bufferHeight - row);
    for (col = 0; col < m_bufferWidth; col++)
    {
      pixelIndex = rowStart + col;
      if (indexMap[pixelIndex] != -1)
      {
        m_buffer->nff.tex[indexMap[pixelIndex]].u = deltaU * (float) col;
        m_buffer->nff.tex[indexMap[pixelIndex]].v = v;
        if (++j >= numVertices)
          break;
      }
    }
    rowStart += m_bufferWidth;
  }
  m_buffer->nff.tr_number = numTriangles;
  free(indexMap);
}

// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - //
// Triangle mesh routines
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - //


// Determine if all vertices of a triangle are associated with good
// data.

static bool
GoodTriangle(BUFFER *buffer, int index1, int index2, int index3)
{
  bool good;

  good = !(buffer->dot[index1].x == 0 && buffer->dot[index1].y == 0 && buffer->dot[index1].z == 0);
  good = good && !(buffer->dot[index2].x == 0 && buffer->dot[index2].y == 0 && buffer->dot[index2].z == 0);
  good = good && !(buffer->dot[index3].x == 0 && buffer->dot[index3].y == 0 && buffer->dot[index3].z == 0);
  return good;
}

// Add a triangle to the mesh

static void
AddTriangle(int index1, int index2, int index3, BUFFER *buffer,
            int *numTriangles, int *indexMap, int *numVertices)
{
  NFF_TR *triangle;

  if (indexMap[index1] == -1)
    indexMap[index1] = (*numVertices)++;

  if (indexMap[index2] == -1)
    indexMap[index2] = (*numVertices)++;

  if (indexMap[index3] == -1)
    indexMap[index3] = (*numVertices)++;

  triangle = &(buffer->nff.triangle[*numTriangles]);

  triangle->vtx1 = indexMap[index1];
  triangle->vtx2 = indexMap[index2];
  triangle->vtx3 = indexMap[index3];
  (*numTriangles)++;
}

// Add the triangles associated with a given quad to the mesh

static void
AddQuadTriangles(QuadtreeNode *quad, BUFFER *buffer, int width,
                 int *numTriangles, int *numVertices, int *indexMap)
{
  int quadVertices[4],goodTriangle;
  PixelCoords lowerRight,upperLeft;

  lowerRight.row = quad->m_lowerLeft.row;
  lowerRight.col = quad->m_upperRight.col;

  upperLeft.col = quad->m_lowerLeft.col;
  upperLeft.row = quad->m_upperRight.row;

  quadVertices[0] = quad->m_lowerLeft.PixelIndex(width);
  quadVertices[1] = lowerRight.PixelIndex(width);
  quadVertices[2] = quad->m_upperRight.PixelIndex(width);
  quadVertices[3] = upperLeft.PixelIndex(width);

  if (quad->m_diagonalIsLeftRight) /* diagonal is lower-left to upper-right */
  {
    /* lower triangle */
    goodTriangle = GoodTriangle(buffer,quadVertices[0],quadVertices[1],
                                quadVertices[2]);
    quad->m_goodSide[BottomSide] = quad->m_goodSide[RightSide] = goodTriangle;
    if (goodTriangle)
      AddTriangle(quadVertices[0],quadVertices[1],quadVertices[2],
                  buffer,numTriangles,indexMap,numVertices);

    /* upper triangle */
    goodTriangle = GoodTriangle(buffer,quadVertices[2],quadVertices[3],
                                quadVertices[0]);
    quad->m_goodSide[LeftSide] = quad->m_goodSide[TopSide] = goodTriangle;
    if (goodTriangle)
      AddTriangle(quadVertices[2],quadVertices[3],quadVertices[0],
                  buffer,numTriangles,indexMap,numVertices);
  }
  else                  /* diagonal is lower-right to upper-left */
  {
    /* lower triangle */
    goodTriangle = GoodTriangle(buffer,quadVertices[0],quadVertices[1],
                                quadVertices[3]);
    quad->m_goodSide[LeftSide] = quad->m_goodSide[BottomSide] = goodTriangle;
    if (goodTriangle)
      AddTriangle(quadVertices[0],quadVertices[1],quadVertices[3],
                  buffer,numTriangles,indexMap,numVertices);

    /* upper triangle */
    goodTriangle = GoodTriangle(buffer,quadVertices[3],quadVertices[1],
                                quadVertices[2]);
    quad->m_goodSide[TopSide] = quad->m_goodSide[RightSide] = goodTriangle;
    if (goodTriangle)
      AddTriangle(quadVertices[3],quadVertices[1],quadVertices[2],
                  buffer,numTriangles,indexMap,numVertices);
  }
}


// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - //
// Routines to align vertices with neighboring quad edges
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - //

// AlignPt aligns point3 with the point closest to it on the line
// defined by point1 & point2

static void
AlignPt(PixelCoords *point1, PixelCoords *point2, PixelCoords *point3,
        int *indexMap, POS3D *vertices, int bufferWidth)
{
  int vert1Index,vert2Index,vert3Index;
  POS3D *pt1,*pt2,*pt3,unitRayVec,rayVec,proj;
  float normalizer,magProj;

  // We check indices in case any of the vertices are from a region of
  // missing or bad data missing data
  if ((vert1Index = indexMap[point1->PixelIndex(bufferWidth)]) < 0)
    return;
  if ((vert2Index = indexMap[point2->PixelIndex(bufferWidth)]) < 0)
    return;
  if ((vert3Index = indexMap[point3->PixelIndex(bufferWidth)]) < 0)
    return;

  if ((vert3Index == vert1Index) || (vert3Index == vert2Index))
  {
    /*     printf("REMARK in AlignPt: vert3index is the same as vert2Index or "
           "vert1Index.\n"); */
    return;
  }
  pt1 = &vertices[vert1Index];
  pt2 = &vertices[vert2Index];
  pt3 = &vertices[vert3Index];

  rayVec.x = pt2->x - pt1->x;
  rayVec.y = pt2->y - pt1->y;
  rayVec.z = pt2->z - pt1->z;

  if ((rayVec.x == 0.0) && (rayVec.y == 0.0) && (rayVec.z == 0.0))
  {
    printf("WARNING in AlignPt: point1 and point2 are geometrically "
           "coincident. (%f %f %f) (%f %f %f)\n",
           pt1->x, pt1->y, pt1->z,
           pt2->x, pt2->y, pt2->z);
    pt3->x = pt1->x;
    pt3->y = pt1->y;
    pt3->z = pt1->z;
    return;
  }

  normalizer = 1.0/sqrt(rayVec.x*rayVec.x +
                        rayVec.y*rayVec.y +
                        rayVec.z*rayVec.z);
  unitRayVec.x = rayVec.x*normalizer;
  unitRayVec.y = rayVec.y*normalizer;
  unitRayVec.z = rayVec.z*normalizer;

  magProj = ((pt3->x - pt1->x)*unitRayVec.x +
             (pt3->y - pt1->y)*unitRayVec.y +
             (pt3->z - pt1->z)*unitRayVec.z);

  proj.x = magProj*unitRayVec.x;
  proj.y = magProj*unitRayVec.y;
  proj.z = magProj*unitRayVec.z;

  pt3->x = pt1->x + proj.x;
  pt3->y = pt1->y + proj.y;
  pt3->z = pt1->z + proj.z;
}


static void
AlignLeafToNeighbor(QuadtreeNode *leaf, QuadtreeNode *neighborLeaf,
                    int side, int *indexMap, POS3D *vertices, int bufferWidth)
{
  PixelCoords upperLeft,lowerRight,nbrUpperLeft,nbrLowerRight;

  if (neighborLeaf == 0)
    return;

  if (leaf == neighborLeaf)
  {
    printf("WARNING in AlignLeafToNeighbor: leaf == neighborLeaf. "
           "Aborting.\n");
    return;
  }

  upperLeft.row = leaf->m_upperRight.row;
  upperLeft.col = leaf->m_lowerLeft.col;
  lowerRight.row = leaf->m_lowerLeft.row;
  lowerRight.col = leaf->m_upperRight.col;

  nbrUpperLeft.row = neighborLeaf->m_upperRight.row;
  nbrUpperLeft.col = neighborLeaf->m_lowerLeft.col;
  nbrLowerRight.row = neighborLeaf->m_lowerLeft.row;
  nbrLowerRight.col = neighborLeaf->m_upperRight.col;

  switch (side)
  {
  case LeftSide:
    AlignPt(&nbrLowerRight,&(neighborLeaf->m_upperRight),
            &(leaf->m_lowerLeft),indexMap,vertices,bufferWidth);
    AlignPt(&nbrLowerRight,&(neighborLeaf->m_upperRight),
            &upperLeft,indexMap,vertices,bufferWidth);
    break;
  case RightSide:
    AlignPt(&(neighborLeaf->m_lowerLeft),&nbrUpperLeft,
            &(leaf->m_upperRight),indexMap,vertices,bufferWidth);
    AlignPt(&(neighborLeaf->m_lowerLeft),&nbrUpperLeft,
            &lowerRight,indexMap,vertices,bufferWidth);
    break;
  case TopSide:
    AlignPt(&(neighborLeaf->m_lowerLeft),&nbrLowerRight,
            &(leaf->m_upperRight),indexMap,vertices,bufferWidth);
    AlignPt(&(neighborLeaf->m_lowerLeft),&nbrLowerRight,
            &upperLeft,indexMap,vertices,bufferWidth);
    break;
  case BottomSide:
    AlignPt(&nbrUpperLeft,&(neighborLeaf->m_upperRight),
            &(leaf->m_lowerLeft),indexMap,vertices,bufferWidth);
    AlignPt(&nbrUpperLeft,&(neighborLeaf->m_upperRight),
            &lowerRight,indexMap,vertices,bufferWidth);
    break;
  }
}

static void
AlignSeamVertices(QuadtreeNode *quadtreeNode, int *indexMap,
                  POS3D *vertices, int width)
{
  list<QuadtreeNode *>::iterator element,endElement;
  QuadtreeNode *leaf;
  QuadtreeNode *neighbor;
  list<QuadtreeNode *> *leaves = new list<QuadtreeNode *>;
//   int numLeaves;                        // debugging
//   int side;                             // debugging

  if (quadtreeNode->IsLeaf())
    return;

  if (!quadtreeNode->HasGrandChildren())
    return;

  quadtreeNode->Leaves(leaves);
//   numLeaves = Leaves(quadtreeNode,leaves); // debugging
//   printf("numLeaves = %d\n",numLeaves);         // debugging

  element = leaves->begin();
  endElement = leaves->end();
  while (!(element == endElement))
  {
    leaf = *element;
    if (leaf->m_neighbors[LeftSide] == 0)
    {
      neighbor = leaf->GetAncestralNeighbor(LeftSide);
      if (neighbor)
      {
//      side = LeftSide; // Debugging...
//      assert(IsAdjacent(leaf,neighbor,&side));
        if (neighbor->m_goodSide[RightSide])
          AlignLeafToNeighbor(leaf,neighbor,LeftSide,indexMap,vertices,width);
      }
    }
    if (leaf->m_neighbors[RightSide] == 0)
    {
      neighbor = leaf->GetAncestralNeighbor(RightSide);
      if (neighbor)
      {
//       side = RightSide; // Debugging...
//       assert(IsAdjacent(leaf,neighbor,&side));
      if (neighbor->m_goodSide[LeftSide])
        AlignLeafToNeighbor(leaf,neighbor,RightSide,indexMap,vertices,width);
      }
    }
    if (leaf->m_neighbors[BottomSide] == 0)
    {
      neighbor = leaf->GetAncestralNeighbor(BottomSide);
      if (neighbor)
      {
//      side = BottomSide; // Debugging...
//      assert(IsAdjacent(leaf,neighbor,&side));
        if (neighbor->m_goodSide[TopSide])
          AlignLeafToNeighbor(leaf,neighbor,BottomSide,indexMap,vertices,width);
      }
    }
    if (leaf->m_neighbors[TopSide] == 0)
    {
      neighbor = leaf->GetAncestralNeighbor(TopSide);
      if (neighbor)
      {
//      side = TopSide; // Debugging...
//      assert(IsAdjacent(leaf,neighbor,&side));
        if (neighbor->m_goodSide[BottomSide])
          AlignLeafToNeighbor(leaf,neighbor,TopSide,indexMap,vertices,width);
      }
    }
    ++element;
  }

  delete leaves;
}

static void
MendTriangleMeshSeams(BUFFER *buffer, int width,
                      int *indexMap, Quadtree *quadtree)
{
  POS3D *vertices = buffer->nff.vtx;

  AlignSeamVertices(quadtree->Root(),indexMap,vertices,width);
}

// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - //
// Main triangle mesh creation routine
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - //


/************************************************************************/
/*                                                                      */
/*                 dot to nff updated version                           */
/*                                                                      */
/************************************************************************/
// #if 0
// static void
// SyntheticTerrain(BUFFER *b, int width, int height)
// {
//   int rowStart,row,col,pixelIndex;
//   pixel *disparities = b->disp;
//   double x,y;
//   double maxZ = -999999.9;
//   /*  double a = 0.2, k = 10.0; works */
//   double a = 1.0, k = 10.0;

//   rowStart = 0;
//   for (row = 0; row < height; row++)
//   {
//     y = (double)row - ((double)height-1.0)/2.0;
//     for (col = 0; col < width; col++)
//     {
//       pixelIndex = rowStart + col;
//       disparities[pixelIndex] = 32;
//       b->dot[pixelIndex].x = x = (double)col - ((double)width-1.0)/2.0;
//       b->dot[pixelIndex].y = y;
//       b->dot[pixelIndex].z = k*(((1.0-exp(-a*x))/(1.0+exp(-a*x))) -
//                              ((1.0-exp(-a*(x-y)))/(1.0+exp(-a*(x-y)))));
//       /*      b->dot[pixelIndex].z = 10.0; */
//       if (b->dot[pixelIndex].z > maxZ)
//      maxZ = b->dot[pixelIndex].z;
//     }
//     rowStart += width;
//   }

//   printf("SyntheticTerrain: maxZ = %e\n",maxZ);
// }
//#endif

static void
Init(BUFFER *b, int width, int height)
{
  b->nff.pt_number = width * height;

  /* Allocate memory for gradient buffer */
  if ((b->gradients =
       (POS2D *) malloc(b->nff.pt_number * sizeof(POS3D))) == NULL)
    {
      fprintf(stderr, "gradient buffer allocation failed.\n");
      exit(EXIT_FAILURE);
    }

  /*  SyntheticTerrain(b,width,height); */
}

static double
Convolve(Array3x3 mask, int row, int col, BUFFER *buffer, int width)
{
  static const int maskRowOrigin = 1;
  static const int maskColOrigin = 1;
  //  pixel *disparities = buffer->disp;
  int i,j,pixelIndex,rowStartIndex;
  double result = 0.0;

  rowStartIndex = (row-maskRowOrigin)*width;
  for (i=0; i < 3; rowStartIndex += width, i++)
    {
      for (j=0; j < 3; j++)
        {
          pixelIndex = rowStartIndex + (col + j-maskColOrigin);
          if (buffer->dot[pixelIndex].x == 0 &&
              buffer->dot[pixelIndex].y == 0 &&
              buffer->dot[pixelIndex].z == 0)
            return (0.0);
//           if ((disparities[pixelIndex] == 0) ||
//               (disparities[pixelIndex] == MISSING_PIXEL) ||
//               (disparities[pixelIndex] == FAR_FIELD_PIXEL))

          result += mask[i][j]*buffer->dot[pixelIndex].z;
        }
    }
  return (result);
}

static void
CalculateGradients(BUFFER *b, int width, int height)
{
  //  pixel *disparities = b->disp;
  int row,col,pixelIndex, numPixels = width*height;
  Array3x3 xMask = {{-1.0/6.0, 0.0, 1.0/6.0},
                    {-1.0/6.0, 0.0, 1.0/6.0},
                    {-1.0/6.0, 0.0, 1.0/6.0}};
  Array3x3 yMask = {{ 1.0/6.0, 1.0/6.0, 1.0/6.0},
                    {   0.0,     0.0,     0.0  },
                    {-1.0/6.0,-1.0/6.0,-1.0/6.0}};

  /* zero out top border row */
  for (pixelIndex = 0; pixelIndex < width; pixelIndex++)
  {
    b->gradients[pixelIndex].x = 0.0;
    b->gradients[pixelIndex].y = 0.0;
  }

  for (row = 1; row < (height-1); row++)
  {
    /* zero out left border column */
    b->gradients[pixelIndex].x = 0.0;
    b->gradients[pixelIndex].y = 0.0;
    pixelIndex++;
    for (col = 1; col < (width - 1); col++)
      {
        if (b->dot[pixelIndex].x == 0 &&
            b->dot[pixelIndex].y == 0 &&
            b->dot[pixelIndex].z == 0)
        {
          b->gradients[pixelIndex].x = Convolve(xMask,row,col,b,width);
          b->gradients[pixelIndex].y = Convolve(yMask,row,col,b,width);
        }
      else
        {
          b->gradients[pixelIndex].x = 0.0;
          b->gradients[pixelIndex].y = 0.0;
        }
      pixelIndex++;
    }
    /* zero out right border column */
    b->gradients[pixelIndex].x = 0.0;
    b->gradients[pixelIndex].y = 0.0;
    pixelIndex++;
  }

  /* zero out bottom border row */
  for (; pixelIndex < numPixels; pixelIndex++)
  {
    b->gradients[pixelIndex].x = 0.0;
    b->gradients[pixelIndex].y = 0.0;
  }
}


// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
// Publicly visible functions
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -

void
dot_to_adaptative_mesh(BUFFER *b, int width, int height, double mesh_tolerance, int max_triangles)
{
  Quadtree *quadtree;

  Init(b,width,height);

  CalculateGradients(b,width,height);
  /*WriteGradientImageFile(b,dft,hd,width,height,'m');*/
  quadtree = new Quadtree(b,width,height);
  printf("\tNumber of triangles in full mesh (%d,%d) = %d\n",
         width,height,2*(width-1)*(height-1));
  printf("\tMesh parameters -->  tol: %f    max triangles: %d\n", mesh_tolerance, max_triangles);
  quadtree->Subdivide(mesh_tolerance, max_triangles);

  printf("Number of triangles in subdivided mesh = %d\n",b->nff.tr_number);
  quadtree->CreateTriangleMesh();

  printf("Number of triangles in reduced mesh = %d\n",b->nff.tr_number);
}

/* original code that place the vertices on a regular grid */
void
dot_to_mesh(BUFFER *b, int width, int height, int h_step, int v_step)
{
  int   y, x, x_grid, y_grid;
  //  int       max_jump = dft->nff_max_jump;
  int   grid_width, grid_height;
  int   buff_pos;
  int   n = 0;                  /* number of triangles created */
  int   AI, BI, CI, DI;         /* shift from pointer to the 4 corners Image */
  int   AG, BG, CG, DG;         /* shift from pointer to the 4 corners Grid */
  double        A1,A2,A3,B1,B2,B3,C1,C2,C3,D1,D2,D3; /* easier to read */
  double distBC,distAD;
#if 0
  double distAB,distAC,distBD,distCD;
  double max_dist;
  double pixel_disp;
  double sin_theta1, sin_theta2;
#endif
  double u,v;

  /* initialize the constants */
  grid_width = (width-1)/h_step+1;
  grid_height = (height-1)/v_step+1;
  b->nff.pt_number = (grid_width) * (grid_height);
  b->nff.tr_number = (grid_width-1)*(grid_height-1)*2;
  x_grid = 0;
  y_grid = 0;

  /* debug */
  printf("w=%d  h=%d\n", width, height);
  printf("sw=%d sh=%d\n", h_step, v_step);
  printf("gw=%d gh=%d\n", grid_width, grid_height);
  printf("pt=%d tr=%d\n", b->nff.pt_number, b->nff.tr_number);


  /* Allocate memory for vertex buffer */
  if ((b->nff.vtx = (POS3D *)malloc(b->nff.pt_number * sizeof(POS3D))) == NULL)
  {
    fprintf(stderr, "nff vertex buffer allocation failed.\n");
    exit(EXIT_FAILURE);
  }
  /* Allocate memory for triangle buffer */
  if ((b->nff.triangle = (NFF_TR *)malloc(b->nff.tr_number * sizeof(NFF_TR))) ==
      NULL)
  {
    fprintf(stderr, "nff triangle buffer allocation failed.\n");
    exit (EXIT_FAILURE);
  }
  /* Allocate memory for texure buffer */
  if ((b->nff.tex = (UV_TEX *)malloc(b->nff.pt_number * sizeof(UV_TEX))) ==
      NULL)
  {
    fprintf(stderr, "nff texure buffer allocation failed.\n");
    exit(EXIT_FAILURE);
  }

  /* make the regular grid point */
  for( y=0, y_grid=0 ; y_grid < grid_height ; y += v_step, y_grid++){
    buff_pos = y_grid*grid_width;
    v = (double)(height-y)/(double)(height);
    for( x=0, x_grid=0 ; x_grid < grid_width ; x += h_step, x_grid ++){
      u = (double)(x)/(double)(width);
      b->nff.vtx[buff_pos+x_grid].x = b->dot[y*width+x].x;
      b->nff.vtx[buff_pos+x_grid].y = b->dot[y*width+x].y;
      b->nff.vtx[buff_pos+x_grid].z = b->dot[y*width+x].z;
      b->nff.tex[buff_pos+x_grid].u = u;
      b->nff.tex[buff_pos+x_grid].v = v;
    }
  }

  // make the triangle
  for( y=0; y < (grid_height-1) ; y ++){
    buff_pos = y*grid_width;
    for( x=0 ; x < (grid_width-1) ; x ++){

// #if 0
//       /* calcul max_dist */
//       pixel_disp = (double)b->disp[y*v_step*width+(x+1)*h_step];
//       sin_theta1 = sin ((double)hd->h_theta_pixel * pixel_disp / 2.0);
//       sin_theta2 = sin ((double)hd->h_theta_pixel* (pixel_disp - max_jump) /
//                      2.0);
//       if(sin_theta1 < THETA_LIMIT)
//         sin_theta1 = THETA_LIMIT;
//       if(sin_theta2 < THETA_LIMIT)
//         sin_theta2 = THETA_LIMIT;
//       max_dist = (double)(dft->baseline/sin_theta1 - dft->baseline/sin_theta2);
//       if (max_dist < 0)
//         max_dist = max_dist *(-1);
// #endif

      /* calcul the distance between the first triangle sommets */
      /* top left corner X */
      AG = buff_pos+x;
      AI = (y)*v_step*width+(x)*h_step;
      A1 = b->nff.vtx[AG].x;
      A2 = b->nff.vtx[AG].y;
      A3 = b->nff.vtx[AG].z;
      /* top right corner Y */
      BG = buff_pos+x+1;
      BI = (y)*v_step*width+(x+1)*h_step;
      B1 = b->nff.vtx[BG].x;
      B2 = b->nff.vtx[BG].y;
      B3 = b->nff.vtx[BG].z;
      /* botom left corner Z */
      CG = buff_pos+x+grid_width;
      CI = (y+1)*v_step*width+(x)*h_step;
      C1 = b->nff.vtx[CG].x;
      C2 = b->nff.vtx[CG].y;
      C3 = b->nff.vtx[CG].z;
      /* botom right corner Y */
      DG = buff_pos+grid_width+x+1;
      DI = (y+1)*v_step*width+(x+1)*h_step;
      D1 = b->nff.vtx[DG].x;
      D2 = b->nff.vtx[DG].y;
      D3 = b->nff.vtx[DG].z;

      distBC = (B1-C1)*(B1-C1) + (B2-C2)*(B2-C2) + (B3-C3)*(B3-C3);
      distAD = (A1-D1)*(A1-D1) + (A2-D2)*(A2-D2) + (A3-D3)*(A3-D3);
#if 0
      distAB = (A1-B1)*(A1-B1) + (A2-B2)*(A2-B2) + (A3-B3)*(A3-B3);
      distAC = (A1-C1)*(A1-C1) + (A2-C2)*(A2-C2) + (A3-C3)*(A3-C3);
      distBD = (B1-D1)*(B1-D1) + (B2-D2)*(B2-D2) + (B3-D3)*(B3-D3);
      distCD = (C1-D1)*(C1-D1) + (C2-D2)*(C2-D2) + (C3-D3)*(C3-D3);
#endif
      /* build the triangles with the smallest diagonal |\| versus |/| */
      if(distBC < distAD){      /* case  |/| */

        /* draw triangle |/ if the jumps in disp is not bigger than allowed */
//      if (!(abs(int(b->disp[AI] - b->disp[BI])) > max_jump ||
//            abs(int(b->disp[BI] - b->disp[CI])) > max_jump ||
//            abs(int(b->disp[CI] - b->disp[DI])) > max_jump ||
//            b->disp[AI] == MISSING_PIXEL ||
//            b->disp[BI] == MISSING_PIXEL ||
//            b->disp[CI] == MISSING_PIXEL)) {
        // Draw a triangle if the vertices are not missing pixels
        if(!((b->dot[AI].x == 0 && b->dot[AI].y ==0 && b->dot[AI].z == 0)  ||
             (b->dot[BI].x == 0 && b->dot[BI].y ==0 && b->dot[BI].z == 0) ||
             (b->dot[CI].x == 0 && b->dot[CI].y ==0 && b->dot[CI].z == 0))) {

          b->nff.triangle[n].vtx1 = AG;
          b->nff.triangle[n].vtx2 = BG;
          b->nff.triangle[n].vtx3 = CG;
          n ++;                 /* you've got one more triangle */
        }
//      /* draw triangle /| if the jumps in disp is not bigger than allowed */
//      if (!(abs(int(b->disp[BI] - b->disp[DI])) > max_jump ||
//            abs(int(b->disp[DI] - b->disp[CI])) > max_jump ||
//            abs(int(b->disp[CI] - b->disp[BI])) > max_jump ||
//            b->disp[BI] == MISSING_PIXEL ||
//            b->disp[CI] == MISSING_PIXEL ||
//            b->disp[DI] == MISSING_PIXEL)) {
        // Draw a triangle if the vertices are not missing pixels
        if(!((b->dot[BI].x == 0 && b->dot[BI].y ==0 && b->dot[BI].z == 0)  ||
             (b->dot[CI].x == 0 && b->dot[CI].y ==0 && b->dot[CI].z == 0) ||
             (b->dot[DI].x == 0 && b->dot[DI].y ==0 && b->dot[DI].z == 0))) {

          b->nff.triangle[n].vtx1 = BG;
          b->nff.triangle[n].vtx2 = DG;
          b->nff.triangle[n].vtx3 = CG;
          n ++;                 /* and a second one */
        }
      }
      else {    /* distBC <= distAD  case |\| */
//      /* draw triangle |\ if the jumps in disp is not bigger than allowed */
//      if (!(abs(int(b->disp[AI] - b->disp[DI])) > max_jump ||
//            abs(int(b->disp[DI] - b->disp[CI])) > max_jump ||
//            abs(int(b->disp[CI] - b->disp[AI])) > max_jump ||
//            b->disp[AI] == MISSING_PIXEL ||
//            b->disp[CI] == MISSING_PIXEL ||
//            b->disp[DI] == MISSING_PIXEL)) {
    // Draw a triangle if the vertices are not missing pixels
    if(!((b->dot[AI].x == 0 && b->dot[AI].y ==0 && b->dot[AI].z == 0)  ||
         (b->dot[CI].x == 0 && b->dot[CI].y ==0 && b->dot[CI].z == 0) ||
         (b->dot[DI].x == 0 && b->dot[DI].y ==0 && b->dot[DI].z == 0))) {

          b->nff.triangle[n].vtx1 = AG;
          b->nff.triangle[n].vtx2 = DG;
          b->nff.triangle[n].vtx3 = CG;
          n ++;                 /* you've got one more triangle */
        }
//      if (!(abs(int(b->disp[BI] - b->disp[DI])) > max_jump ||
//            abs(int(b->disp[DI] - b->disp[AI])) > max_jump ||
//            abs(int(b->disp[AI] - b->disp[BI])) > max_jump ||
//            b->disp[AI] == MISSING_PIXEL ||
//            b->disp[BI] == MISSING_PIXEL ||
//            b->disp[DI] == MISSING_PIXEL)) {
    // Draw a triangle if the vertices are not missing pixels
    if(!((b->dot[AI].x == 0 && b->dot[AI].y ==0 && b->dot[AI].z == 0)  ||
         (b->dot[BI].x == 0 && b->dot[BI].y ==0 && b->dot[BI].z == 0) ||
         (b->dot[DI].x == 0 && b->dot[DI].y ==0 && b->dot[DI].z == 0))) {
          b->nff.triangle[n].vtx1 = BG;
          b->nff.triangle[n].vtx2 = DG;
          b->nff.triangle[n].vtx3 = AG;
          n ++;                 /* and a second one */
        }
      }
    }
  }
  b->nff.tr_number = n;

  return;
}

/***********************/
/* write inventor_file */
/***********************/

#if defined(ASP_HAVE_PKG_OPENSCENEGRAPH) && ASP_HAVE_PKG_OPENSCENEGRAPH==1
void write_osg_impl(BUFFER *b, std::string const& filename,
                         std::string const& texture_filename) {

  std::cout << "Writing " << filename << "..." << std::flush;

  // A geode is a "geometry node". It is-a 'Node' and contains 'Drawable's.
  osg::ref_ptr<osg::Geode> geode (new osg::Geode());

  // 'Geometry' is-a 'Drawable'. It is a collection of vertices, normals,
  // colors, texture coordinates and so on. It is organized in "primitive
  // sets", that allow to say that, e.g., "from vertex to 0 to 8 render as
  // triangles, from 9 to 13 render as points, please". For those
  // OpenGL-inclined, think of 'Geometry' as a wrapper around vertex (and
  // normals, and texcoord) arrays and 'glDrawElements()'
  osg::ref_ptr<osg::Geometry> geometry (new osg::Geometry());

  // Create and set the vertex array for the geometry object
  osg::ref_ptr<osg::Vec3Array> vertices (new osg::Vec3Array());

  for (int i = 0; i < b->nff.pt_number; ++i)
    vertices->push_back (osg::Vec3 (b->nff.vtx[i].x, b->nff.vtx[i].y, b->nff.vtx[i].z) );

  geometry->setVertexArray (vertices.get());


  std::vector<vw::Vector3> vertex_normals(b->nff.pt_number);

  // Now create the "primitive set", which describes which vertices
  // are to be used when rendering the triangles.
  osg::ref_ptr<osg::DrawElementsUInt> faces(new osg::DrawElementsUInt(osg::PrimitiveSet::TRIANGLES, 0));
  for (int i = 0; i < b->nff.tr_number; ++i) {
    faces->push_back(b->nff.triangle[i].vtx1);
    faces->push_back(b->nff.triangle[i].vtx2);
    faces->push_back(b->nff.triangle[i].vtx3);
    vw::Vector3 v1(b->nff.vtx[b->nff.triangle[i].vtx1].x,
                   b->nff.vtx[b->nff.triangle[i].vtx1].y,
                   b->nff.vtx[b->nff.triangle[i].vtx1].z);
    vw::Vector3 v2(b->nff.vtx[b->nff.triangle[i].vtx2].x,
                   b->nff.vtx[b->nff.triangle[i].vtx2].y,
                   b->nff.vtx[b->nff.triangle[i].vtx2].z);
    vw::Vector3 v3(b->nff.vtx[b->nff.triangle[i].vtx3].x,
                   b->nff.vtx[b->nff.triangle[i].vtx3].y,
                   b->nff.vtx[b->nff.triangle[i].vtx3].z);
    vw::Vector3 l1 = v3-v1;
    vw::Vector3 l2 = v3-v2;
    vw::Vector3 c = normalize(vw::math::cross_prod(l1,l2));
    vertex_normals[b->nff.triangle[i].vtx1] = c;
    vertex_normals[b->nff.triangle[i].vtx2] = c;
    vertex_normals[b->nff.triangle[i].vtx3] = c;
  }

  // Create an array for the single normal.
  osg::ref_ptr<osg::Vec3Array> n = new osg::Vec3Array;
  geometry->setNormalArray( n.get() );
  for (int i = 0; i < b->nff.pt_number; ++i)
    n->push_back( osg::Vec3( vertex_normals[i].x(),
                             vertex_normals[i].y(),
                             vertex_normals[i].z()) );

  geometry->addPrimitiveSet(faces.get());

  // Next, we associate some texture coordinates with the vertices
  osg::Vec2Array* texcoords = new osg::Vec2Array(b->nff.pt_number);
  for (int i = 0; i < b->nff.pt_number; ++i)
    (*texcoords)[i].set(b->nff.tex[i].u, b->nff.tex[i].v);
  geometry->setTexCoordArray(0,texcoords);

  // The geometry is now full specified, so we associate it with the geode.
  geode->addDrawable (geometry.get());

  // We must now associate the texture in a file with the geode
  osg::ref_ptr<osg::Texture2D> texture(new osg::Texture2D);

  // protect from being optimized away as static state
  texture->setDataVariance(osg::Object::DYNAMIC);
  texture->setUseHardwareMipMapGeneration(false);

  osg::Image* texture_image = osgDB::readImageFile(texture_filename.c_str());
  if (!texture_image)
    throw vw::ArgumentErr() << "OpenSceneGraph couldn't read the texture, " << texture_filename << ".";
  texture->setImage(texture_image);

  // Create a new StateSet with default settings:
  osg::ref_ptr<osg::StateSet> texture_state_set(new osg::StateSet());

  // Assign texture unit 0 of our new StateSet to the texture
  // we just created and enable the texture.
  texture_state_set->setTextureAttributeAndModes(0,texture.get(),osg::StateAttribute::ON);
  geode->setStateSet(texture_state_set.get());

  // Set up lighting properties
  osg::StateSet* state = geode->getOrCreateStateSet();
  osg::ref_ptr<osg::Material> mat = new osg::Material;
  mat->setDiffuse( osg::Material::FRONT_AND_BACK,
                   osg::Vec4( .7f, .7f, .7f, 1.f ) );
  mat->setSpecular( osg::Material::FRONT_AND_BACK,
                    osg::Vec4( .1f, .1f, .1f, 1.f ) );
  mat->setShininess( osg::Material::FRONT_AND_BACK, 20.f );
  state->setAttribute( mat.get() );

  // Add the geometry to the geode and save the geode to a file o disk.
  osgDB::writeNodeFile(*(geode.get()), filename);

  std::cout << " done.\n";
}

#else  // HAVE_PKG_OPENSCENEGRAPH

void write_osg_impl(BUFFER *b, std::string const& filename,
                         std::string const& texture_filename) {
  std::cout << "WARNING: could not write " << filename << ".  Compiled without open scene graph support.";
}
#endif  // HAVE_PKG_OPENSCENEGRAPH


void write_inventor_impl(BUFFER *b, std::string const& filename,
                         std::string const& texture_filename,
                         bool flip_triangles) {
  FILE *outflow;
  int i;

  std::vector<std::string> texture_path_components;
  boost::split( texture_path_components, texture_filename, boost::is_any_of("/") );
  std::string relative_texture_path = "./" + texture_path_components[texture_path_components.size()-1];

  // open output file
  if((outflow = fopen (filename.c_str(), "w" )) == 0) {
    fprintf (stderr, "write_inventor: cannot open output file: %s\n", filename.c_str());
    exit(EXIT_FAILURE);
  }
  fprintf (outflow, "#Inventor V2.0 ascii\n\n");
  fprintf (outflow, "# Created by the Intelligent Robotics Group,\n");
  fprintf (outflow, "# NASA Ames Research Center\n");
  fprintf (outflow, "# File generated by the NASA Ames Stereo-Pipeline.\n");
  fprintf (outflow, "# %d vertices, %d triangles\n\n", b->nff.pt_number, b->nff.tr_number);
  fprintf (outflow, "Separator {\n");
  fprintf (outflow, "   Texture2 {\n");

  // Grab the end of the full path
  fprintf (outflow, "  filename \"%s\"\n", relative_texture_path.c_str());

  fprintf (outflow, "   }\n");
  // Material Section
  fprintf (outflow, "Material {\n");
  fprintf (outflow, "  ambientColor 1.00 1.00 1.00\n");
  fprintf (outflow, "  diffuseColor 1.00 1.00 1.00\n");
  fprintf (outflow, "  specularColor 0.00 0.00 0.00\n");
  fprintf (outflow, "  emissiveColor 0.00 0.00 0.00\n");
  fprintf (outflow, "  shininess 0.00\n");
  fprintf (outflow, "  transparency 0.00\n");
  fprintf (outflow, "}\n");

  // Gouraud shading (comment out in the file, but make it easy to
  // turn on if desired.
  if (flip_triangles) {
    fprintf (outflow, "ShapeHints {\n");
    fprintf (outflow, "   vertexOrdering CLOCKWISE\n");
    fprintf (outflow, "#  creaseAngle 3.1\n");
    fprintf (outflow, "}\n");
  } else {
    fprintf (outflow, "#ShapeHints {\n");
    fprintf (outflow, "#  creaseAngle 3.1\n");
    fprintf (outflow, "#}\n");
  }

  // vertices position
  fprintf (outflow, "    Coordinate3 {\n");
  fprintf (outflow, "        point [    ");
  fprintf (outflow, "# %d vertices\n", b->nff.pt_number);
  for(i = 0 ; i < b->nff.pt_number; i++ )
    fprintf (outflow, "%f %f %f,\n",
             b->nff.vtx[i].x, b->nff.vtx[i].y, b->nff.vtx[i].z);
  fprintf (outflow, "                ]\n");
  fprintf (outflow, "    }\n");

  // uv coordinate
  fprintf (outflow, "    TextureCoordinate2 {\n");
  fprintf (outflow, "        point [    ");
  for(i = 0 ; i < b->nff.pt_number; i++ )
    fprintf (outflow, "%f %f,\n",  b->nff.tex[i].u, b->nff.tex[i].v);
  fprintf (outflow, "                ]\n");
  fprintf (outflow, "    }\n");

  // polygons
  fprintf (outflow, "    IndexedFaceSet {\n");
  fprintf (outflow, "        coordIndex [     \n");
  fprintf (outflow, "# %d triangles\n", b->nff.tr_number);

  for(i = 0 ; i < b->nff.tr_number; i++ )
    fprintf (outflow, "%d, %d, %d, -1,\n", b->nff.triangle[i].vtx1,
             b->nff.triangle[i].vtx3, b->nff.triangle[i].vtx2);

  fprintf (outflow, "        ]\n");
  fprintf (outflow, "    }\n");
  fprintf (outflow, "}\n");

#if 0
  fprintf (outflow, "    ShapeHints {\n");
  fprintf (outflow, "        vertexOrdering  COUNTERCLOCKWISE\n");
  fprintf (outflow, "        creaseAngle     1.0\n");
  fprintf (outflow, "    }\n");
  fprintf (outflow, "    Material {\n");
  fprintf (outflow, "        ambientColor    0.2 0.2 0.2\n");
  fprintf (outflow, "        diffuseColor    0.42 0.41 0.35\n");
  fprintf (outflow, "        specularColor   0.2 0.2 0.2\n");
  fprintf (outflow, "        emissiveColor   0 0 0\n");
  fprintf (outflow, "        shininess       0\n");
  fprintf (outflow, "        transparency    0.5\n");
  fprintf (outflow, "    }\n");
  fprintf (outflow, "#   Texture2 {\n");
  fprintf (outflow, "#      filename \"tex2.rgb\"\n");
  fprintf (outflow, "#       wrapS   REPEAT\n");
  fprintf (outflow, "#       wrapT   REPEAT\n");
  fprintf (outflow, "#       model   MODULATE\n");
  fprintf (outflow, "#   }\n");
  fprintf (outflow, "#   Texture2Transform {\n");
  fprintf (outflow, "#       rotation        0.0\n");
  fprintf (outflow, "#       center          0.5 0.5\n");
  fprintf (outflow, "#       translation     0.0 0.0\n");
  fprintf (outflow, "#       scaleFactor     1.0 1.0\n");
  fprintf (outflow, "#   }\n");
  fprintf (outflow, "    Coordinate3 {\n");
  fprintf (outflow, "        point [    ");
  fprintf (outflow, "# %d vertices\n", b->nff.pt_number);

  for(i = 0 ; i < b->nff.pt_number; i++ )
    fprintf (outflow, "%f %f %f,\n", b->nff.vtx[i].x, b->nff.vtx[i].y,
             b->nff.vtx[i].z);

  fprintf (outflow, "                ]\n");
  fprintf (outflow, "    }\n");
  fprintf (outflow, "    IndexedFaceSet {\n");
  fprintf (outflow, "        coordIndex [     \n");
  fprintf (outflow, "# %d triangles\n", b->nff.tr_number);

  for(i = 0 ; i < b->nff.tr_number; i++ )
    fprintf (outflow, "%d, %d, %d, -1,\n", b->nff.triangle[i].vtx1,
             b->nff.triangle[i].vtx2, b->nff.triangle[i].vtx3);

  fprintf (outflow, "        ]\n");
  fprintf (outflow, "    }\n");
  fprintf (outflow, "}\n");

#endif

  printf("%s writen successfully\n", filename.c_str());

  fclose(outflow);
} // write_inventor_file

/*******************/
/* write vrml_file */
/*******************/

void write_vrml_impl(BUFFER *b, std::string const& filename, std::string const& texture_filename) {
  FILE *outflow;
  int i;

  // open output file
  if((outflow = fopen (filename.c_str(), "w" )) == 0) {
    fprintf (stderr, "write_vrml(): cannot open output file: %s\n", filename.c_str());
    exit(EXIT_FAILURE);
  }
  fprintf(outflow,"#VRML V1.0 ascii\n#\n");
  fprintf(outflow,"# Created by the Intelligent Robotics Group,\n");
  fprintf(outflow,"# NASA Ames Research Center\n");
  fprintf(outflow,"# File generated by the NASA Ames Stereo-Pipeline.\n");
  fprintf(outflow,"# %d vertices, %d triangles\n\n", b->nff.pt_number, b->nff.tr_number);

  fprintf (outflow, "Separator {\n");

  //   fprintf (outflow, "RotationXYZ {\n");
  //   fprintf (outflow, "axis X\n");
  //   fprintf (outflow, "angle 3.14159265358979\n");
  //   fprintf (outflow, "}\n");

  // Disabled for now until we have a better mechanism for passing in
  // these settings. -mbroxton
  //
  //   fprintf (outflow, "Material {\n");
  //   fprintf (outflow, "  ambientColor  %4.2f %4.2f %4.2f \n", dft->ambiColorRed,
  //       dft->ambiColorGreen, dft->ambiColorBlue );
  //   fprintf (outflow, "  diffuseColor  %4.2f %4.2f %4.2f \n", dft->diffColorRed,
  //       dft->diffColorGreen, dft->diffColorBlue );
  //   fprintf (outflow, "  specularColor  %4.2f %4.2f %4.2f \n", dft->specColorRed,
  //       dft->specColorGreen, dft->specColorBlue );
  //   fprintf (outflow, "  emissiveColor  %4.2f %4.2f %4.2f \n", dft->emisColorRed,
  //       dft->emisColorGreen, dft->emisColorBlue );
  //   fprintf (outflow, "  shininess %4.2f \n", dft->shininess);
  //   fprintf (outflow, "  transparency %4.2f \n}\n\n", dft->transparency);

  fprintf (outflow, "ShapeHints {\n");
  fprintf (outflow, "  vertexOrdering COUNTERCLOCKWISE\n");
//   if(dft->shapeType_solid)
    fprintf (outflow, "  shapeType SOLID\n");
//   else
//     fprintf (outflow, "  shapeType UNKNOWN_SHAPE_TYPE\n");
  fprintf (outflow, "  creaseAngle 3.1\n}\n\n");

  /* vertices position */
  fprintf (outflow, "Coordinate3 { \n");
  fprintf (outflow, "  point [\n");
  for(i = 0 ; i < b->nff.pt_number; i++ )
    fprintf (outflow, "    %f %f %f,\n", b->nff.vtx[i].x,
             b->nff.vtx[i].y, b->nff.vtx[i].z);
  fprintf (outflow, "    ]\n");
  fprintf (outflow, "}\n\n");

  // Texture file
  fprintf (outflow, "Texture2 {\n");
  fprintf (outflow, "  filename \"%s\"\n", texture_filename.c_str());
  fprintf (outflow, "}\n\n");

  /* uv coordinate */
  fprintf (outflow, "TextureCoordinate2 {\n");
  fprintf (outflow, "  point [\n");
  for(i = 0 ; i < b->nff.pt_number; i++ )
    fprintf (outflow, "    %f %f,\n",  b->nff.tex[i].u, b->nff.tex[i].v);
  fprintf (outflow, "    ]\n");
  fprintf (outflow, "}\n\n");

  /* polygons */
  fprintf (outflow, "IndexedFaceSet {\n");
  fprintf (outflow, "  coordIndex [\n");
  for(i = 0 ; i < b->nff.tr_number; i++ )
    fprintf (outflow, "    %d, %d, %d, -1,\n", b->nff.triangle[i].vtx1,
             b->nff.triangle[i].vtx3, b->nff.triangle[i].vtx2);
  fprintf (outflow, "    ]\n\n");
  fprintf (outflow, "  textureCoordIndex [\n");
  for(i = 0 ; i < b->nff.tr_number; i++ )
    fprintf (outflow, "    %d, %d, %d, -1,\n", b->nff.triangle[i].vtx1,
             b->nff.triangle[i].vtx3, b->nff.triangle[i].vtx2);
  fprintf (outflow, "    ]\n");
  fprintf (outflow, "}\n");

  // Close off Separator
  fprintf (outflow, "}\n");

  printf("%s written successfully\n", filename.c_str());
  fclose(outflow);
} // write_vrml_file

void write_trimesh_impl(BUFFER *b, std::string const& filename, bool flip_triangles) {
  FILE *outflow;
  int i;

  // open output file
  if((outflow = fopen (filename.c_str(), "w" )) == 0) {
    fprintf (stderr, "write_inventor: cannot open output file: %s\n", filename.c_str());
    exit(EXIT_FAILURE);
  }

  // vertices
  fprintf (outflow, "%d\n", b->nff.pt_number);

  for(i = 0 ; i < b->nff.pt_number; i++ )
    fprintf (outflow, "%f %f %f\n", b->nff.vtx[i].x,
             b->nff.vtx[i].y, b->nff.vtx[i].z);

  // triangles
  fprintf (outflow, "%d\n", b->nff.tr_number);

  for(i = 0 ; i < b->nff.tr_number; i++ ) {
                if (flip_triangles) {
                        fprintf (outflow, "%d %d %d\n", b->nff.triangle[i].vtx1,
                                                         b->nff.triangle[i].vtx2, b->nff.triangle[i].vtx3);
                } else {
                        fprintf (outflow, "%d %d %d\n", b->nff.triangle[i].vtx1,
                                                         b->nff.triangle[i].vtx3, b->nff.triangle[i].vtx2);
                }
        }

} // write_trimesh_file
